
#include <hip/hip_runtime.h>
#ifdef OSQP
#include "../head/ADMMGPUConstCons.cuh"

ADMMGPUConstCons::ADMMGPUConstCons() : MethodP2P()
{
#if DEBUG_CONSTRUCTOR
	std::cout << "Constructeur ADMMGPUConstCons" << std::endl;
#endif // DEBUG_CONSTRUCTOR
	_name = NAME;
}
/// <summary>
///  TOOOOOOOOOOOO  DOOOOOOOOOOOOOOOOOOOOOOOOOOO    bp3 !!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
/// et ne pas faire l'opf lorsqu'il n'y a pas de contraintes de r�seaux !!!!!!!!!!!!!!!!!!!
/// </summary>
/// <param name="rho"></param>

ADMMGPUConstCons::ADMMGPUConstCons(float rho) : MethodP2P()
{
#if DEBUG_CONSTRUCTOR
	std::cout << "Constructeur ADMMGPUConstCons defaut" << std::endl;
#endif // DEBUG_CONSTRUCTOR
	_name = NAME;
	_rho = rho;
}

ADMMGPUConstCons::~ADMMGPUConstCons()
{
	
	DELETEA(Q);
	DELETEA(xResult);
	
}

void ADMMGPUConstCons::setParam(float rho)
{
	_rho = rho;
}

void ADMMGPUConstCons::setTau(float tau)
{
	if (tau < 1) {
		throw std::invalid_argument("tau must be greater than 1");
	}
	_tau = tau;
}

void ADMMGPUConstCons::init(const Simparam& sim, const StudyCase& cas)
{
	// intitilisation des matrixs et variables 
	_rhog = sim.getRho();
	_rho1 = sim.getRho1();
	
	//std::cout << "rho initial " << _rhog << std::endl;
	_nAgent = sim.getNAgent();
	
	_rhol = _rho;
	if (_rho == 0) {
		_rhol = _rhog;
	}
	const int iterG = sim.getIterG();
	const int stepG = sim.getStepG();
	float epsG = sim.getEpsG();
	float epsGC = sim.getEpsGC();
	_ratioEps = epsG / epsGC;
	
	nVoisinCPU = cas.getNvoi();
	nVoisin = MatrixGPU(nVoisinCPU, 1);

	int nVoisinMax = nVoisin.max2();
	if (_blockSize * NMAXPEERPERTRHREAD < nVoisinMax) {
		std::cout << _blockSize << " " << NMAXPEERPERTRHREAD << " " << nVoisinMax << std::endl;
		throw std::invalid_argument("For this Method, an agent must not have more than 5120 peers");
	}

	_nLine = cas.getNLine();
		
	_nBus = cas.getNBus();

	_nTrade = nVoisin.sum();
	_numBlocksN = ceil((_nAgent + _blockSize - 1) / _blockSize);
	_numBlocksM = ceil((_nTrade + _blockSize - 1) / _blockSize);
	_numBlocksL = ceil((_nLine + _blockSize - 1) / _blockSize);
	_numBlocksNL = ceil((_nAgent * _nLine + _blockSize - 1) / _blockSize);
	_at1 = _rhog; // represente en fait 2*a
	_at2 = _rhol;

	resF = MatrixCPU(3, (iterG / stepG) + 1);
	resX = MatrixCPU(4, (iterG / stepG) + 1);

	MatrixCPU BETA(cas.getBeta());
	MatrixGPU Ub(cas.getUb());
	MatrixGPU Lb(cas.getLb());
	LAMBDA = sim.getLambda();
	trade = sim.getTrade();
	
	//std::cout << "mise sous forme lin�aire" << std::endl;
	// Rem : si matrice d�j� existante, elles sont d�j� sur GPU donc bug pour les get

	CoresMatLin = MatrixGPU(_nAgent, _nAgent, -1);
	CoresLinAgent = MatrixGPU(_nTrade, 1);
	CoresAgentLin = MatrixGPU(_nAgent + 1, 1);
	CoresLinVoisin = MatrixGPU(_nTrade, 1);
	CoresLinTrans = MatrixGPU(_nTrade, 1);
	
	Tlocal_pre = MatrixGPU(_nTrade, 1);
	tradeLin = MatrixGPU(_nTrade, 1);
	LAMBDALin = MatrixGPU(_nTrade, 1);

	matLb = MatrixGPU(_nTrade, 1);
	matUb = MatrixGPU(_nTrade, 1);
	Ct = MatrixGPU(_nTrade, 1);

	int indice = 0;

	for (int idAgent = 0; idAgent < _nAgent; idAgent++) {
		MatrixCPU omega(cas.getVoisin(idAgent));
		int Nvoisinmax = nVoisinCPU.get(idAgent, 0);
		for (int voisin = 0; voisin < Nvoisinmax; voisin++) {
			int idVoisin = omega.get(voisin, 0);
			if(Lb.getNCol()==1){
				matLb.set(indice, 0, Lb.get(idAgent, 0));
				matUb.set(indice, 0, Ub.get(idAgent, 0));
			} else {
				matLb.set(indice, 0, Lb.get(idAgent, idVoisin));
				matUb.set(indice, 0, Ub.get(idAgent, idVoisin));
			}
			Ct.set(indice, 0, BETA.get(idAgent, idVoisin));
			tradeLin.set(indice, 0, trade.get(idAgent, idVoisin));
			Tlocal_pre.set(indice, 0, trade.get(idAgent, idVoisin));
			LAMBDALin.set(indice, 0, LAMBDA.get(idAgent, idVoisin));
			CoresLinAgent.set(indice, 0, idAgent);
			CoresLinVoisin.set(indice, 0, idVoisin);
			CoresMatLin.set(idAgent, idVoisin, indice);
			indice = indice + 1;
		}
		CoresAgentLin.set(idAgent + 1, 0, indice);
	}
	for (int lin = 0; lin < _nTrade; lin++) {
		int i = CoresLinAgent.get(lin, 0);
		int j = CoresLinVoisin.get(lin, 0);
		int k = CoresMatLin.get(j, i);
		CoresLinTrans.set(lin, 0, k);
	}


	// transfert des mises lineaire
	matUb.transferGPU();
	matLb.transferGPU();
	Ct.transferGPU();

	Tlocal_pre.transferGPU();
	tradeLin.transferGPU();
	LAMBDALin.transferGPU();

	CoresAgentLin.transferGPU();
	CoresLinAgent.transferGPU();
	CoresLinVoisin.transferGPU();
	CoresMatLin.transferGPU();
	CoresLinTrans.transferGPU();
	
	
#ifndef OSQPGPU
	//std::cout << "donnees sur GPU pour le grid" << std::endl;
	if (_nLine) {	
		settings = new OSQPSettings;
		work = new OSQPWorkspace;
		data = new OSQPData;
		if (settings) {
			osqp_set_default_settings(settings);
			settings->alpha = 1;
			settings->eps_abs = sim.getEpsL()/5;
			settings->eps_rel = 0;
			settings->verbose = 0;
			settings->max_iter = sim.getIterL();
			settings->adaptive_rho_interval = 0;
		}
		lLimit = MatrixGPU(cas.getLineLimit());
		PsoCPU = MatrixCPU(_nAgent, 1);
		Q = new c_float[_nAgent];
		
		c_float* U = new c_float[_nLine + 1];
		c_float* L = new c_float[_nLine + 1];
		xResult = new c_float[_nAgent];
		for (int l = 0; l < _nLine; l++) {

			L[l] = -lLimit.get(l, 0);
			U[l] =  lLimit.get(l, 0);
			
		}
		L[_nLine] = 0;
		U[_nLine] = 0;
		qTosqp = sim.getPn(); // somme des trades
			
		for (int n = 0; n < _nAgent; n++) {
			Q[n] = _rhog * qTosqp.get(n, 0) / 2;
			xResult[n] = qTosqp.get(n, 0);
		}
		Aosqp = MatrixCPU(_nLine + 1, _nAgent, 1);
		MatrixCPU temp(cas.getPowerSensi());
		Aosqp.setBloc(0, _nLine, 0, _nAgent, &temp);
		//Aosqp = cas.getPowerSensi();


		Hosqp = MatrixCPU(_nAgent, _nAgent);
		Hosqp.setEyes(_rho1);
			
		c_int H_nnz = Hosqp.getNNullHalf();
		int Hcol = Hosqp.getNCol();
			
		c_float* Hdata = new c_float[H_nnz];
		c_int* Hidx = new c_int[H_nnz];
		c_int* Hptr = new c_int[Hcol + 1];
		Hosqp.toCSCHalf(Hdata, Hidx, Hptr);

			
		//Aosqp.display();
		c_int A_nnz = Aosqp.getNNull();
		int Acol = Aosqp.getNCol();
		c_float* Adata = new c_float[A_nnz];
		c_int* Aidx = new c_int[A_nnz];
		c_int* Aptr = new c_int[Acol + 1];
		Aosqp.toCSC(Adata, Aidx, Aptr);
			
		data = new OSQPData;
		if (data) {
			data->n = _nAgent;
			//data->m = _nLine;
			data->m = _nLine + 1;
			data->P = csc_matrix(data->n, data->n, H_nnz, Hdata, Hidx, Hptr);
			data->q = Q;
			data->A = csc_matrix(data->m, data->n, A_nnz, Adata, Aidx, Aptr);
			data->l = L;
			data->u = U;
		}
		c_int exitflag = osqp_setup(&work, data, settings);

		osqp_warm_start_x(work, xResult);
		DELETEA(L);
		DELETEA(U);

		DELETEA(Hdata);
		DELETEA(Hidx);
		DELETEA(Hptr);
		DELETEA(Adata);
		DELETEA(Aidx);
		DELETEA(Aptr);
	}
#endif
	Pso = MatrixGPU(_nAgent, 1, 0, 1); // = Pn ? risque de non respect des contraintes
	
	etaP = MatrixGPU(_nAgent, 1, 0, 1); 

	//std::cout << "autres donn�e sur GPU" << std::endl;
	tempNN = MatrixGPU(_nTrade, 1, 0, 1);
	tempN1 = MatrixGPU(_nAgent, 1, 0, 1); // plut�t que de re-allouer de la m�moire � chaque utilisation
	tempL1 = MatrixGPU(_nLine, 1, 0, 1);
	tempL2 = MatrixGPU(_nLine, 1, 0, 1);
	//MatrixGPU temp1N(1, _nAgent, 0, 1);

	Tlocal = MatrixGPU(_nTrade, 1, 0, 1);
	P = MatrixGPU(_nAgent, 1, 0, 1); // moyenne des trades
	Pn = MatrixGPU(sim.getPn(), 1); // somme des trades


	a = MatrixGPU(cas.geta(), 1);
	b = MatrixGPU(cas.getb(), 1);
	Ap2 = a;
	Ap1 = nVoisin;
	Ap3 = nVoisin;
	Ap123 = MatrixGPU(_nAgent, 1, 0, 1);
	Bp3 = MatrixGPU(_nAgent, 1, 0, 1); // 1/Mn * (Pso + P)/2 - eta/rho1

	Bt1 = MatrixGPU(_nTrade, 1, 0, 1);
	Cp = b;

	Pmin = MatrixGPU(cas.getPmin(), 1);
	Pmax = MatrixGPU(cas.getPmax(), 1);
	MU = MatrixGPU(sim.getMU(), 1); // facteur reduit i.e lambda_l/_rho
	Tmoy = MatrixGPU(sim.getPn(), 1);

	tempNN.preallocateReduction();
	Tlocal.preallocateReduction();
	tempL1.preallocateReduction();
	

	P.preallocateReduction();
	Pso.preallocateReduction();


	Pmin.divideT(&nVoisin);
	Pmax.divideT(&nVoisin);
	Ap1.multiply(_rhol);
	Ap3.multiplyT(&nVoisin);
	Ap3.multiply(_rho1);
	Cp.multiplyT(&nVoisin);
	Tmoy.divideT(&nVoisin);

	Ap2.multiplyT(&nVoisin);
	Ap2.multiplyT(&nVoisin);
	Ap123.add(&Ap1, &Ap2);
	Ap123.add(&Ap3);
		
	updateGlobalProbGPU();

	//Hosqp.display();
	
	//std::cout << " end init " << std::endl;
}



void ADMMGPUConstCons::updateP0(const StudyCase& cas)
{
	_id = _id + 1;
#ifdef INSTRUMENTATION
	cudaDeviceSynchronize();
	std::chrono::high_resolution_clock::time_point t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION

	matLb.transferCPU();
	
	Pmin = MatrixGPU(cas.getPmin());
	Pmax = MatrixGPU(cas.getPmax());


	MatrixGPU Lb(cas.getLb());

	b = cas.getb();
	Cp = cas.getb();
	int indice = 0;

	for (int idAgent = 0; idAgent < _nAgent; idAgent++) {
		int Nvoisinmax = nVoisinCPU.get(idAgent, 0);
		for (int voisin = 0; voisin < Nvoisinmax; voisin++) {
			matLb.set(indice, 0, Lb.get(idAgent, 0));
			indice = indice + 1;
		}
	}

	
	matLb.transferGPU();

	Pmin.divideT(&nVoisin);
	Pmax.divideT(&nVoisin);
	Cp.multiplyT(&nVoisin);
#ifdef INSTRUMENTATION
	cudaDeviceSynchronize();
	std::chrono::high_resolution_clock::time_point t2 = std::chrono::high_resolution_clock::now();
	timePerBlock.increment(0, 10, (float) std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
	occurencePerBlock.increment(0, 10, 1);
#endif // INSTRUMENTATION

	//std::cout << "fin update temps : " << (float)(clock() - t) / CLOCKS_PER_SEC << std::endl;
}



void ADMMGPUConstCons::solveOPF()
{
	
	updateQ();
#ifndef OSQPGPU
	osqp_update_lin_cost(work, Q);
	
	// FB 1b
	osqp_solve(work);
	xResult = work->solution->x;
#endif
	for (int n = 0; n < _nAgent; n++) {
		PsoCPU.set(n, 0, xResult[n]);
	}
	Pso = PsoCPU; // 2eme transfert

}



void ADMMGPUConstCons::updateQ()
{
	tempN1.add(&Pn, &Pso);
	tempN1.multiply(-_rho1 / 2);
	tempN1.add(&etaP);
	tempN1.toMatCPU(qTosqp); // 1 er transfert
	
	//std::cout << " qT :" << std::endl;
	//qTosqp.display();
	
	for (int n = 0; n < _nAgent; n++) {
		Q[n] = qTosqp.get(n, 0);
	}
}


void ADMMGPUConstCons::solve(Simparam* result, const Simparam& sim, const StudyCase& cas)
{
#ifdef DEBUG_SOLVE
	cas.display();
	sim.display(1);
#endif // DEBUG_SOLVE
	
	
	clock_t tall = clock();
#ifdef INSTRUMENTATION
	std::chrono::high_resolution_clock::time_point t1;
	std::chrono::high_resolution_clock::time_point t2;
#endif // INSTRUMENTATION

	if (_id == 0) {
#ifdef INSTRUMENTATION
		cudaDeviceSynchronize();
		t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION
		init(sim, cas);
#ifdef INSTRUMENTATION
		cudaDeviceSynchronize();
		t2 = std::chrono::high_resolution_clock::now();
		timePerBlock.increment(0, 0, (float) std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
		occurencePerBlock.increment(0, 0, 1);
#endif // INSTRUMENTATION
	}
	//std::cout << _numBlocks2 << " " <<  _blockSize << std::endl;
	
	_at1 = _rhog; // represente en fait 2*a
	
	float epsG = sim.getEpsG();
	float epsL = sim.getEpsL();
	const int stepL = sim.getStepL();
	const int stepG = sim.getStepG();
	const int iterG = sim.getIterG();
	const int iterL = sim.getIterL();
	

	float resG = 2 * epsG;
	float epsL2 = epsL * epsL;
	int iterGlobal = 0;
	int iterLocal = 0;
	int realOccurence = 0;
	
	//std::cout << iterG << " " << iterL << " " << epsL << " " << epsG << std::endl;
	while ((iterGlobal < iterG) && (resG > epsG)) {
#ifdef INSTRUMENTATION
		cudaDeviceSynchronize();
		t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION

		updateLocalProbGPU(epsL2, iterL);
#ifdef INSTRUMENTATION
		cudaDeviceSynchronize();
		t2 = std::chrono::high_resolution_clock::now();
		timePerBlock.increment(0, 1, (float) std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
#endif // INSTRUMENTATION

		tradeLin.swap(&Tlocal); // echange juste les pointeurs	
		//std::cout << "-";
		
		updateGlobalProbGPU();
		
		if (!(iterGlobal % stepG)) {
#ifdef INSTRUMENTATION
			cudaDeviceSynchronize();
			t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION
			resG = updateRes(&resF, &Tlocal, iterGlobal / stepG, &tempNN);
#ifdef INSTRUMENTATION
			cudaDeviceSynchronize();
			t2 = std::chrono::high_resolution_clock::now();
			timePerBlock.increment(0, 8, (float) std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
#endif // INSTRUMENTATION
		}
		//std::cout << iterGlobal << " " << iterLocal << " " << resL << " " << resF.get(0, iterGlobal / stepG) << " " << resF.get(1, iterGlobal / stepG) << std::endl;
		iterGlobal++;
	}
#ifdef INSTRUMENTATION
	occurencePerBlock.increment(0, 1, iterGlobal);
	occurencePerBlock.increment(0, 5, iterGlobal);
	occurencePerBlock.increment(0, 6, iterGlobal);
	occurencePerBlock.increment(0, 7, iterGlobal);
	occurencePerBlock.increment(0, 8, iterGlobal / stepG);

	cudaDeviceSynchronize();
	t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION
	
	

	float fc = calcFc();
	//std::cout << iterGlobal << " " << iterLocal << " " << resL << " " << resG << std::endl;
	MatrixCPU tradeLinCPU;
	tradeLin.toMatCPU(tradeLinCPU);
	MatrixCPU LAMBDALinCPU;
	LAMBDALin.toMatCPU(LAMBDALinCPU);
	MatrixCPU PnCPU;
	Pn.toMatCPU(PnCPU);
	MatrixCPU MUCPU;
	MU.toMatCPU(MUCPU);
	

	int indice = 0;
	for (int idAgent = 0;idAgent < _nAgent; idAgent++) {
		MatrixCPU omega(cas.getVoisin(idAgent));
		int Nvoisinmax = nVoisinCPU.get(idAgent, 0);
		for (int voisin = 0; voisin < Nvoisinmax; voisin++) {
			int idVoisin = omega.get(voisin, 0);
			trade.set(idAgent, idVoisin, tradeLinCPU.get(indice, 0));
			LAMBDA.set(idAgent, idVoisin, LAMBDALinCPU.get(indice, 0));
			indice = indice + 1;
		}
	}
	result->setResF(&resF);
	result->setLAMBDA(&LAMBDA);
	result->setTrade(&trade);
	
	result->setIter(iterGlobal);
	result->setPn(&PnCPU);
	result->setFc(fc);
	result->setMU(&MUCPU);
	result->setRho(_rhog);
#ifdef INSTRUMENTATION
	cudaDeviceSynchronize();
	t2 = std::chrono::high_resolution_clock::now();
	timePerBlock.increment(0, 9, (float) std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
	occurencePerBlock.increment(0, 9, 1);
	result->setTimeBloc(&timePerBlock, &occurencePerBlock);
#endif // INSTRUMENTATION

	tall = clock() - tall;
	result->setTime((float)tall / CLOCKS_PER_SEC);
}

void ADMMGPUConstCons::updateLocalProbGPU(float epsL, int nIterL) {
	int numBlocks = _nAgent;
	switch (_blockSize) {
	case 512:
		updateTradePGPUSharedResidualCons<512> << <numBlocks, _blockSize >> > (Tlocal._matrixGPU, Tlocal_pre._matrixGPU, Tmoy._matrixGPU, P._matrixGPU, MU._matrixGPU, nVoisin._matrixGPU, _at1, _at2, Bt1._matrixGPU, Ct._matrixGPU,
			matLb._matrixGPU, matUb._matrixGPU, Ap1._matrixGPU, Ap3._matrixGPU, Ap123._matrixGPU, Bp3._matrixGPU, Cp._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, CoresAgentLin._matrixGPU, epsL, nIterL);
		break;
	case 256:
		updateTradePGPUSharedResidualCons<256> << <numBlocks, _blockSize >> > (Tlocal._matrixGPU, Tlocal_pre._matrixGPU, Tmoy._matrixGPU, P._matrixGPU, MU._matrixGPU, nVoisin._matrixGPU, _at1, _at2, Bt1._matrixGPU, Ct._matrixGPU,
			matLb._matrixGPU, matUb._matrixGPU, Ap1._matrixGPU, Ap3._matrixGPU, Ap123._matrixGPU, Bp3._matrixGPU, Cp._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, CoresAgentLin._matrixGPU, epsL, nIterL);
		break;
	case 128:
		updateTradePGPUSharedResidualCons<128> << <numBlocks, _blockSize >> > (Tlocal._matrixGPU, Tlocal_pre._matrixGPU, Tmoy._matrixGPU, P._matrixGPU, MU._matrixGPU, nVoisin._matrixGPU, _at1, _at2, Bt1._matrixGPU, Ct._matrixGPU,
			matLb._matrixGPU, matUb._matrixGPU, Ap1._matrixGPU, Ap3._matrixGPU, Ap123._matrixGPU, Bp3._matrixGPU, Cp._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, CoresAgentLin._matrixGPU, epsL, nIterL);
		break;
	case 64:
		updateTradePGPUSharedResidualCons< 64> << <numBlocks, _blockSize >> > (Tlocal._matrixGPU, Tlocal_pre._matrixGPU, Tmoy._matrixGPU, P._matrixGPU, MU._matrixGPU, nVoisin._matrixGPU, _at1, _at2, Bt1._matrixGPU, Ct._matrixGPU,
			matLb._matrixGPU, matUb._matrixGPU, Ap1._matrixGPU, Ap3._matrixGPU, Ap123._matrixGPU, Bp3._matrixGPU, Cp._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, CoresAgentLin._matrixGPU, epsL, nIterL);
		break;
	case 32:
		updateTradePGPUSharedResidualCons< 32> << <numBlocks, _blockSize >> > (Tlocal._matrixGPU, Tlocal_pre._matrixGPU, Tmoy._matrixGPU, P._matrixGPU, MU._matrixGPU, nVoisin._matrixGPU, _at1, _at2, Bt1._matrixGPU, Ct._matrixGPU,
			matLb._matrixGPU, matUb._matrixGPU, Ap1._matrixGPU, Ap3._matrixGPU, Ap123._matrixGPU, Bp3._matrixGPU, Cp._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, CoresAgentLin._matrixGPU, epsL, nIterL);
		break;
	case 16:
		updateTradePGPUSharedResidualCons< 16> << <numBlocks, _blockSize >> > (Tlocal._matrixGPU, Tlocal_pre._matrixGPU, Tmoy._matrixGPU, P._matrixGPU, MU._matrixGPU, nVoisin._matrixGPU, _at1, _at2, Bt1._matrixGPU, Ct._matrixGPU,
			matLb._matrixGPU, matUb._matrixGPU, Ap1._matrixGPU, Ap3._matrixGPU, Ap123._matrixGPU, Bp3._matrixGPU, Cp._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, CoresAgentLin._matrixGPU, epsL, nIterL);
		break;
	case  8:
		updateTradePGPUSharedResidualCons<  8> << <numBlocks, _blockSize >> > (Tlocal._matrixGPU, Tlocal_pre._matrixGPU, Tmoy._matrixGPU, P._matrixGPU, MU._matrixGPU, nVoisin._matrixGPU, _at1, _at2, Bt1._matrixGPU, Ct._matrixGPU,
			matLb._matrixGPU, matUb._matrixGPU, Ap1._matrixGPU, Ap3._matrixGPU, Ap123._matrixGPU, Bp3._matrixGPU, Cp._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, CoresAgentLin._matrixGPU, epsL, nIterL);
		break;
	case  4:
		updateTradePGPUSharedResidualCons<  4> << <numBlocks, _blockSize >> > (Tlocal._matrixGPU, Tlocal_pre._matrixGPU, Tmoy._matrixGPU, P._matrixGPU, MU._matrixGPU, nVoisin._matrixGPU, _at1, _at2, Bt1._matrixGPU, Ct._matrixGPU,
			matLb._matrixGPU, matUb._matrixGPU, Ap1._matrixGPU, Ap3._matrixGPU, Ap123._matrixGPU, Bp3._matrixGPU, Cp._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, CoresAgentLin._matrixGPU, epsL, nIterL);
		break;
	case  2:
		updateTradePGPUSharedResidualCons<  2> << <numBlocks, _blockSize >> > (Tlocal._matrixGPU, Tlocal_pre._matrixGPU, Tmoy._matrixGPU, P._matrixGPU, MU._matrixGPU, nVoisin._matrixGPU, _at1, _at2, Bt1._matrixGPU, Ct._matrixGPU,
			matLb._matrixGPU, matUb._matrixGPU, Ap1._matrixGPU, Ap3._matrixGPU, Ap123._matrixGPU, Bp3._matrixGPU, Cp._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, CoresAgentLin._matrixGPU, epsL, nIterL);
		break;
	case  1:
		updateTradePGPUSharedResidualCons<  1> << <numBlocks, _blockSize >> > (Tlocal._matrixGPU, Tlocal_pre._matrixGPU, Tmoy._matrixGPU, P._matrixGPU, MU._matrixGPU, nVoisin._matrixGPU, _at1, _at2, Bt1._matrixGPU, Ct._matrixGPU,
			matLb._matrixGPU, matUb._matrixGPU, Ap1._matrixGPU, Ap3._matrixGPU, Ap123._matrixGPU, Bp3._matrixGPU, Cp._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, CoresAgentLin._matrixGPU, epsL, nIterL);
		break;
	}
	//cudaStreamSynchronize(streamCalculation);
}



void ADMMGPUConstCons::updateGlobalProbGPU()
{
	//Rem : tout calcul qui est de taille N ou M peut �tre fait par les agents
		// Si le calcul est de taile L, soit c'est calcul� par un/des superviseurs, soit tous les agents le calcul (un peu absurde)

#ifdef INSTRUMENTATION
// FB 3a
	cudaDeviceSynchronize();
	std::chrono::high_resolution_clock::time_point t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION


	updatePnGPU << <_numBlocksN, _blockSize >> > (Pn._matrixGPU, Tmoy._matrixGPU, nVoisin._matrixGPU, _nAgent);
	
#ifdef INSTRUMENTATION
	cudaDeviceSynchronize();
	std::chrono::high_resolution_clock::time_point t2 = std::chrono::high_resolution_clock::now();

	timePerBlock.increment(0, 5, (float) std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());

	// FB 3b
	cudaDeviceSynchronize();
	t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION	

	// Resolution de l'OPF
	if (_nLine) {
		/*std::cout << " Pn :" << std::endl;
		Pn.display(true);
		std::cout << " etaP :" << std::endl;
		etaP.display(true);*/
		solveOPF();
		/*std::cout << " Pso :" << std::endl;
		PsoCPU.display();
		std::cout << " ----------------" << std::endl;*/
		//Pso.display(true);
	}
	else {
		Pso = Pn;
	}
	
	
	

#ifdef INSTRUMENTATION
	cudaDeviceSynchronize();
	t2 = std::chrono::high_resolution_clock::now();
	timePerBlock.increment(0, 6, (float) std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());

	// FB 3c
	cudaDeviceSynchronize();
	t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION

	// update Bp3
	updateEtaPBp3 << <_numBlocksN, _blockSize >> > (Bp3._matrixGPU, etaP._matrixGPU, nVoisin._matrixGPU, Pso._matrixGPU, Pn._matrixGPU, _rho1, _nAgent);
	updateLAMBDABt1GPU << <_numBlocksM, _blockSize >> > (Bt1._matrixGPU, LAMBDALin._matrixGPU, tradeLin._matrixGPU, _rhog, CoresLinTrans._matrixGPU, _nTrade);
	

#ifdef INSTRUMENTATION
	cudaDeviceSynchronize();
	t2 = std::chrono::high_resolution_clock::now();
	timePerBlock.increment(0, 7, (float) std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
#endif // INSTRUMENTATION
}



float ADMMGPUConstCons::updateRes(MatrixCPU* res, MatrixGPU* Tlocal, int iter, MatrixGPU* tempNN)
{

	float resS = Tlocal->max2(&tradeLin);

	updateDiffGPU <<<_numBlocksM, _blockSize >> > (tempNN->_matrixGPU, Tlocal->_matrixGPU, CoresLinTrans._matrixGPU, _nAgent);
	float resR = tempNN->max2();

	float resXf = _ratioEps * Pso.max2(&Pn);
	res->set(0, iter, resR);
	res->set(1, iter, resS);
	res->set(2, iter, resXf);
	return MYMAX(MYMAX(resXf, resS), resR);

}



void ADMMGPUConstCons::display() {

	std::cout << _name << std::endl;
}

#endif





