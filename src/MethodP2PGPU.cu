#include "hip/hip_runtime.h"
#include "../head/MethodP2PGPU.cuh"
 


MethodP2PGPU::MethodP2PGPU() : Method(){

}
MethodP2PGPU::~MethodP2PGPU(){

}


float MethodP2PGPU::updateRes(int iter)
{
	float resS = Tlocal.max2(&tradeLin);
	updateDiffGPU << <_numBlocksM, _blockSize >> > (tempNN._matrixGPU, tradeLin._matrixGPU, CoresLinTrans._matrixGPU, _nTrade);
	float resR = tempNN.max2();

	updateResX << <_numBlocksL, _blockSize >> > (tempL1._matrixGPU, Kappa1._matrixGPU, Kappa2._matrixGPU, Kappa1_pre._matrixGPU, Kappa2_pre._matrixGPU, _nLine);

	resF.set(0, iter, resR);
	resF.set(1, iter, resS);
	if (iter > 0 && _tau > 1) {
		if (resR > _mu * resS) {
			_rhog = _tau * _rhog;
			_at1 = _rhog;
			//std::cout << iter << ", rho augmente :" << _rhog << std::endl;
		}
		else if (resS > _mu * resR) {// rho = rho / tau_inc;
			_rhog = _rhog / _tau;
			_at1 = _rhog;
			//std::cout << iter << ", rho diminue :" << _rhog << std::endl;
		}
	}
	
	return MYMAX(resS, resR);
}
float MethodP2PGPU::updateResEndo(int iter)
{
	float resS = Tlocal.max2(&tradeLin);
	updateDiffGPU << <_numBlocksM, _blockSize >> > (tempNN._matrixGPU, tradeLin._matrixGPU, CoresLinTrans._matrixGPU, _nTrade);
	float resR = tempNN.max2();

	updateResX << <_numBlocksL, _blockSize >> > (tempL1._matrixGPU, Kappa1._matrixGPU, Kappa2._matrixGPU, Kappa1_pre._matrixGPU, Kappa2_pre._matrixGPU, _nLine);


	float resXf = _ratioEps * sqrt(tempL1.max2());
	

	resF.set(0, iter, resR);
	resF.set(1, iter, resS);
	resF.set(2, iter, resXf);
	
	return MYMAX(MYMAX(resXf, resS), resR);
}
float MethodP2PGPU::calcRes()
{
	float d1 = Tlocal.max2(&Tlocal_pre);
	float d2 = P.max2(&Tmoy);

	return d1* (d1 > d2) + d2 * (d2 >= d1);
}


void MethodP2PGPU::updateLAMBDA(MatrixGPU* LAMBDA, MatrixGPU* trade, float rho, MatrixGPU* tempNN)
{
	tempNN->set(trade);
	tempNN->addTrans(trade);
	tempNN->multiply(rho);
	tempNN->multiply(0.5);
	LAMBDA->add(LAMBDA, tempNN);
}
void MethodP2PGPU::updateKappa()
{
	Kappa1.projectNeg();
	Kappa1.add(&lLimit);
	Kappa1.subtract(&Qtot);
	Kappa2.projectNeg();
	Kappa2.add(&lLimit);
	Kappa2.add(&Qtot);
}
void MethodP2PGPU::updatePn()
{
	Pn.set(&Tmoy);
	Pn.multiplyT(&nVoisin);
}


void MethodP2PGPU::updateP0(const StudyCase& cas)
{
	_id = _id + 1;
#ifdef INSTRUMENTATION
	hipDeviceSynchronize();
	std::chrono::high_resolution_clock::time_point t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION
	// Change : Power Limits, cost function


	MatrixGPU Lb(cas.getLb());
	MatrixGPU Ub(cas.getUb());
	MatrixCPU BETA(cas.getBeta());

	matLb.transferCPU();
	matUb.transferCPU();
	Ct.transferCPU();
	CoresLinVoisin.transferCPU();

	if (cas.isAC() && !isAC) {
		MatrixGPU aT(cas.geta(), 1);
		MatrixGPU bT(cas.getb(), 1);
		MatrixGPU PminT(cas.getPmin(), 1);
		MatrixGPU PmaxT(cas.getPmax(), 1);

		a.setFromBloc(0, _nAgent, 0, 1, &aT);
		b.setFromBloc(0, _nAgent, 0, 1, &bT);
		Pmin.setFromBloc(0, _nAgent, 0, 1, &PminT);
		Pmax.setFromBloc(0, _nAgent, 0, 1, &PmaxT);
	}
	else if ((!cas.isAC()) && isAC){
		throw std::invalid_argument("updateP0 : Study Case is not AC, but this method require AC information");
	}
	else {
		a = cas.geta();
		b = cas.getb();
		Pmin = cas.getPmin();
		Pmax = cas.getPmax();
	}
	Cp1 = b;
	int indice = 0;

	// hypothese : ce sont les mêmes voisins !!!
	for (int idAgent = 0; idAgent < _nAgent; idAgent++) {
		int Nvoisinmax = (int) nVoisinCPU.get(idAgent, 0);
		for (int voisin = 0; voisin < Nvoisinmax; voisin++) {
			int idVoisin = (int) CoresLinVoisin.get(indice, 0);
			if(Lb.getNCol()==1){
				matLb.set(indice, 0, Lb.get(idAgent, 0));
				matUb.set(indice, 0, Ub.get(idAgent, 0));
			} else {
				matLb.set(indice, 0, Lb.get(idAgent, idVoisin));
				matUb.set(indice, 0, Ub.get(idAgent, idVoisin));
			}
			Ct.set(indice, 0, BETA.get(idAgent, idVoisin));
			indice = indice + 1;
		}
	}
	for (int idAgent = _nAgentTrue; idAgent < _nAgent; idAgent++) {
		for (int voisin = 0; voisin < (_nAgent - 1); voisin++) {
			int idVoisin = (int) CoresLinVoisin.get(indice, 0);
			if(Lb.getNCol()==1){
				matLb.set(indice, 0, Lb.get(idAgent, 0));
				matUb.set(indice, 0, Ub.get(idAgent, 0));
			} else {
				matLb.set(indice, 0, Lb.get(idAgent, idVoisin));
				matUb.set(indice, 0, Ub.get(idAgent, idVoisin));
			}
			Ct.set(indice, 0, BETA.get(idAgent, idVoisin));
			indice = indice + 1;
		}
	}


	matLb.transferGPU();
	matUb.transferGPU();
	Ct.transferGPU();
	CoresLinVoisin.transferGPU();

	Pmin.divideT(&nVoisin);
	Pmax.divideT(&nVoisin);
	Cp1.multiplyT(&nVoisin);

	Ap2a = a;
	Ap2.add(&Ap2a, &Ap2b);
	Ap2.multiplyT(&nVoisin);
	Ap2.multiplyT(&nVoisin);
	Ap12.add(&Ap1, &Ap2);
	Ap123.add(&Ap12, &Ap3);
	Cp.add(&Cp1, &Cp2);

#ifdef INSTRUMENTATION
	hipDeviceSynchronize();
	std::chrono::high_resolution_clock::time_point t2 = std::chrono::high_resolution_clock::now();
	timePerBlock.increment(0, 8, (float) std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
	occurencePerBlock.increment(0, 8, 1);
#endif // INSTRUMENTATION
	
}

void MethodP2PGPU::initLinForm( const Simparam& sim, const StudyCase& cas){

	MatrixCPU BETA(cas.getBeta());
	MatrixCPU Ub(cas.getUb());
	MatrixCPU Lb(cas.getLb());
	LAMBDA = sim.getLambda(); 
	trade = sim.getTrade();

	// Rem : si matrice deja existante, elles sont deja sur GPU donc bug pour les get

	if (Ct.getPos()) { // une copie en trop mais pour l'instant c'est ok...
		CoresMatLin.transferCPU();

		CoresLinAgent.transferCPU();
		CoresAgentLin.transferCPU();
		CoresLinVoisin.transferCPU();
		CoresLinTrans.transferCPU();

		Tlocal_pre.transferCPU();
		tradeLin.transferCPU();
		LAMBDALin.transferCPU();

		matLb.transferCPU();
		matUb.transferCPU();
		Ct.transferCPU();

	}

	CoresMatLin = MatrixGPU(_nAgent, _nAgentTrue, -1);
	CoresAgentLin = MatrixGPU(_nAgent + 1, 1);
	CoresLinAgent = MatrixGPU(_nTrade, 1);
	CoresLinVoisin = MatrixGPU(_nTrade, 1);
	CoresLinTrans = MatrixGPU(_nTrade, 1);

	Tlocal_pre = MatrixGPU(_nTrade, 1);
	tradeLin = MatrixGPU(_nTrade, 1);
	LAMBDALin = MatrixGPU(_nTrade, 1);

	matLb = MatrixGPU(_nTrade, 1);
	matUb = MatrixGPU(_nTrade, 1);
	Ct = MatrixGPU(_nTrade, 1);
	

	int indice = 0;
	//std::cout << " P " << std::endl;
	for (int idAgent = 0; idAgent < _nAgentTrue; idAgent++) { // P
		MatrixCPU omega(cas.getVoisin(idAgent));
		int Nvoisinmax = (int) nVoisinCPU.get(idAgent, 0);
		for (int voisin = 0; voisin < Nvoisinmax; voisin++) {
			int idVoisin = (int) omega.get(voisin, 0);
			if(Lb.getNCol()==1){
				matLb.set(indice, 0, Lb.get(idAgent, 0));
				matUb.set(indice, 0, Ub.get(idAgent, 0));
			} else {
				matLb.set(indice, 0, Lb.get(idAgent, idVoisin));
				matUb.set(indice, 0, Ub.get(idAgent, idVoisin));
			}
			Ct.set(indice, 0, BETA.get(idAgent, idVoisin));
			tradeLin.set(indice, 0, trade.get(idAgent, idVoisin));
			Tlocal_pre.set(indice, 0, trade.get(idAgent, idVoisin));
			LAMBDALin.set(indice, 0, LAMBDA.get(idAgent, idVoisin));
			CoresLinAgent.set(indice, 0, idAgent);
			CoresLinVoisin.set(indice, 0, idVoisin);
			CoresMatLin.set(idAgent, idVoisin, indice);
			indice = indice + 1;
		}
		CoresAgentLin.set(idAgent + 1, 0, indice);
	}
	//std::cout << " Q " << std::endl;
	for (int idAgent = _nAgentTrue; idAgent < _nAgent; idAgent++) { // Q
		for (int idVoisin = 0; idVoisin < _nAgentTrue; idVoisin++) {
			if (idVoisin != (idAgent - _nAgentTrue)) {
				if(Lb.getNCol()==1){
					matLb.set(indice, 0, Lb.get(idAgent, 0));
					matUb.set(indice, 0, Ub.get(idAgent, 0));
				} else {
					matLb.set(indice, 0, Lb.get(idAgent, idVoisin));
					matUb.set(indice, 0, Ub.get(idAgent, idVoisin));
				}
				tradeLin.set(indice, 0, trade.get(idAgent, idVoisin));
				Tlocal_pre.set(indice, 0, trade.get(idAgent, idVoisin));
				LAMBDALin.set(indice, 0, LAMBDA.get(idAgent, idVoisin));
				CoresLinAgent.set(indice, 0, idAgent);
				CoresLinVoisin.set(indice, 0, idVoisin + _nAgentTrue);
				CoresMatLin.set(idAgent, idVoisin, indice);
				indice = indice + 1;
			}
		}

		CoresAgentLin.set(idAgent + 1, 0, indice);
	}
	for (int lin = 0; lin < _nTrade; lin++) {
		int i = (int) CoresLinAgent.get(lin, 0);
		int j = (int) CoresLinVoisin.get(lin, 0);
		if (lin >= _nTradeP) {
			i -= _nAgentTrue;
		}

		int k = (int) CoresMatLin.get(j, i);
		CoresLinTrans.set(lin, 0, k);
	}

		
	// transfert des mises lineaire
	matUb.transferGPU();
	matLb.transferGPU();
	Ct.transferGPU();

	Tlocal_pre.transferGPU();
	tradeLin.transferGPU();
	LAMBDALin.transferGPU();

	CoresAgentLin.transferGPU();
	CoresLinAgent.transferGPU();
	CoresLinVoisin.transferGPU();
	CoresMatLin.transferGPU();
	CoresLinTrans.transferGPU();
}

void MethodP2PGPU::initSize(const StudyCase& cas){
	_nAgentTrue = cas.getNagent();
	_nAgent = _nAgentTrue + isAC * _nAgentTrue;
	if (cas.isAC() && !isAC) {
		MatrixCPU nVoisinT = cas.getNvoi();
		nVoisinCPU = MatrixCPU(_nAgent, 1);
		for (int n = 0; n < _nAgent; n++) {
			nVoisinCPU.set(n, 0, nVoisinT.get(n, 0));
		}
	}else if(!cas.isAC() && isAC){
		throw std::invalid_argument("initSize : Study Case is not AC, but this method require AC information");
	}
	else {
		nVoisinCPU = cas.getNvoi();
	}
	nVoisin = MatrixGPU(nVoisinCPU, 1);
	nVoisin.preallocateReduction();

	_nLine = cas.getNLine();
	_nBus = cas.getNBus();
	_nTrade = (int) nVoisin.sum();
	_nTradeP = 0;
	if(!isAC){
		_nTradeP = _nTrade;
		_nTradeQ = 0;
	} else{
		for (int n = 0; n < _nAgentTrue; n++) {
			_nTradeP += (int) nVoisin.get(n, 0);
		}
		_nTradeQ = _nTrade - _nTradeP;
	}
	_numBlocksN = ceil((_nAgent + _blockSize - 1) / _blockSize);
	_numBlocksM = ceil((_nTrade + _blockSize - 1) / _blockSize);
	_numBlocksL = ceil((_nLine + _blockSize - 1) / _blockSize);
	_numBlocksNL = ceil((_nAgent*_nLine + _blockSize - 1) / _blockSize);


}

void MethodP2PGPU::initSimParam(const Simparam& sim){
	
	_rhog = sim.getRho();
	_rho1 = sim.getRho1();
	_rhol = _rho;
	if (_rho == 0) {
		_rhol = _rhog;
	}

	_iterG = sim.getIterG();
	_iterL = sim.getIterL();
	_iterIntern = sim.getIterIntern();

	_stepG = sim.getStepG();
	_stepL = sim.getStepL();
	_stepIntern = sim.getStepIntern();

	_epsG = sim.getEpsG();
	_epsX = sim.getEpsGC();
	_epsIntern = sim.getEpsIntern();
	_epsL = sim.getEpsL();
	_ratioEps = _epsG / _epsX;

	resF = MatrixCPU(3, (_iterG / _stepG) + 1);
	resX = MatrixCPU(4, (_iterG / _stepG) + 1);

	tempNN = MatrixGPU(_nTrade, 1, 0, 1);
	tempN1 = MatrixGPU(_nAgent, 1, 0, 1); // plut�t que de re-allouer de la m�moire � chaque utilisation
	tempL1 = MatrixGPU(_nLine, 1, 0, 1);
	tempL2 = MatrixGPU(_nLine, 1, 0, 1);

	
	tempNN.preallocateReduction();
	tempL1.preallocateReduction();

}

void MethodP2PGPU::initDCEndoGrid(const StudyCase& cas){
	
	Kappa1 = MatrixGPU(_nLine, 1, 0, 1);
	Kappa2 = MatrixGPU(_nLine, 1, 0, 1);
	Kappa1_pre = MatrixGPU(_nLine, 1, 0, 1);
	Kappa2_pre = MatrixGPU(_nLine, 1, 0, 1);
	Qpart = MatrixGPU(_nAgent, _nLine, 0, 1);
	Qtot = MatrixGPU(_nLine, 1, 0, 1);
	alpha = MatrixGPU(_nAgent, _nLine, 0, 1);

	G = MatrixGPU(cas.getPowerSensi());

	lLimit = MatrixGPU(cas.getLineLimit(), 1);

	GTrans = MatrixGPU(_nAgent, _nLine);
	if (GTrans.getPos()) {
		GTrans.transferCPU();
		G.transferCPU();
	}


	GTrans.setTrans(&G);

	G.transferGPU();
	GTrans.transferGPU();


	G2 = GTrans;
	G2.multiplyT(&GTrans);

}

void MethodP2PGPU::initDCEndoMarket(){
	initP2PMarket();

	Ap2a = a;
	Ap2b = MatrixGPU(_nAgent, 1, 0, 1);
	Ap3 = MatrixGPU(_nAgent, 1, 0, 1); // not used by default but exists
	Ap123 = MatrixGPU(_nAgent, 1, 0, 1); // idem

	Cp2 = MatrixGPU(_nAgent, 1, 0, 1);
	Cp1 = b;

	Cp1.multiplyT(&nVoisin);
	

	Ap2b.sum(&G2);
	Ap2b.multiply(2 * _rho1);
	Ap2.add(&Ap2a, &Ap2b);

	Ap2.multiplyT(&nVoisin);
	Ap2.multiplyT(&nVoisin);
	Ap12.add(&Ap1, &Ap2);
	Cp = Cp1;
}
void MethodP2PGPU::initP2PMarket(){
	_at1 = _rhog; 
	_at2 = _rhol;
	Ap2 = a;
	Ap1 = nVoisin;
	Ap12 = MatrixGPU(_nAgent, 1, 0, 1);

	Bt1 = MatrixGPU(_nTrade, 1, 0, 1);
	Cp = b;

	
	Pmin.divideT(&nVoisin);
	Pmax.divideT(&nVoisin);
	Ap1.multiply(_rhol);
	Cp.multiplyT(&nVoisin);
	Tmoy.divideT(&nVoisin);
	
	Ap2.multiplyT(&nVoisin);
	Ap2.multiplyT(&nVoisin);
	Ap12.add(&Ap1, &Ap2);

}

void MethodP2PGPU::initCaseParam(const Simparam& sim,const StudyCase& cas){

	Tlocal = MatrixGPU(_nTrade, 1, 0, 1);
	P = MatrixGPU(_nAgent, 1, 0, 1); // moyenne des trades
	Pn = MatrixGPU(_nAgent, 1, 0, 1); // somme des trades

	// si cas AC, a, b , Nvoisin, Pmin, Pmax n'ont pas la bonne taille !!!
	if (cas.isAC() && !isAC) {
		MatrixGPU aT(cas.geta(), 1);
		MatrixGPU bT(cas.getb(), 1);
		MatrixGPU PminT(cas.getPmin(), 1);
		MatrixGPU PmaxT(cas.getPmax(), 1);
		MatrixGPU MUT(sim.getMU(), 1); // facteur reduit i.e lambda_l/_rho
		MatrixGPU TmoyT(sim.getPn(), 1);
		a = MatrixGPU(_nAgent, 1, 0, 1);
		b = MatrixGPU(_nAgent, 1, 0, 1);
		Pmin = MatrixGPU(_nAgent, 1, 0, 1);
		Pmax = MatrixGPU(_nAgent, 1, 0, 1);
		MU = MatrixGPU(_nAgent, 1, 0, 1);
		Tmoy = MatrixGPU(_nAgent, 1, 0, 1);

		
		a.setFromBloc(0, _nAgent, 0, 1, &aT);
		b.setFromBloc(0, _nAgent, 0, 1, &bT);
		Pmax.setFromBloc(0, _nAgent, 0, 1, &PmaxT);
		Pmin.setFromBloc(0, _nAgent, 0, 1, &Pmin);
		MU.setFromBloc(0, _nAgent, 0, 1, &MUT);
		Tmoy.setFromBloc(0, _nAgent, 0, 1, &TmoyT);
	}
	else if(!cas.isAC() && isAC){
		throw std::invalid_argument("initCaseParam : Study Case is not AC, but this method require AC information");
	}
	else {
		a = MatrixGPU(cas.geta(), 1);
		b = MatrixGPU(cas.getb(), 1);

		Pmin = MatrixGPU(cas.getPmin(), 1);
		Pmax = MatrixGPU(cas.getPmax(), 1);
		MU = MatrixGPU(sim.getMU(), 1); // facteur reduit i.e lambda_l/_rho
		Tmoy = MatrixGPU(sim.getPn(), 1);
	}
	Pn = Tmoy;

	Tlocal.preallocateReduction();
	P.preallocateReduction();
}



void MethodP2PGPU::solveWithMinPower(Simparam* result, const Simparam& sim, const StudyCase& cas)
{
	std::cout << "solveWithMinPower : should not be called" << std::endl;
}




float MethodP2PGPU::calcFc()
{
	
	tempN1.set(&a);
	
	tempN1.multiply(0.5);
	tempN1.multiplyT(&Pn);
	
	tempN1.add(&b);
	
	tempN1.multiplyT(&Pn);
	
	float fc = tempN1.sum();
	

	tempNN.set(&trade);
	
	tempNN.multiplyT(&Ct);
	
	fc = fc + tempNN.sum();



	//std::cout << "fc " << fc << std::endl;
	return fc;

}


void MethodP2PGPU::display(){
	std::cout << " resolution par la methode " << _name << std::endl;
}



__global__ void updateLAMBDAGPU(float* LAMBDALin, float* tradeLin, float rho, float* CoresLinTrans, int const N)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int step = blockDim.x * gridDim.x;
	for (int l = index; l < N; l += step)
	{
		float m = LAMBDALin[l];
		int k = CoresLinTrans[l];
		LAMBDALin[l] = m + 0.5 * rho * (tradeLin[l] + tradeLin[k]);
	}
}
__global__ void updateBt1GPU(float* Bt1, float* tradeLin, float rho, float* LAMBDA, float* CoresLinTrans, int const N)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int step = blockDim.x * gridDim.x;
	for (int l = index; l < N; l += step)
	{
		int k = CoresLinTrans[l];
		Bt1[l] = 0.5 * (tradeLin[l] - tradeLin[k]) - LAMBDA[l] / rho;
	}

}

__global__ void updateLAMBDABt1GPU(float* Bt1, float* LAMBDA, float* tradeLin, float rho, float* CoresLinTrans, int const N) {

	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int step = blockDim.x * gridDim.x;
	for (int l = index; l < N; l += step)
	{
		int k = CoresLinTrans[l];
		float m = LAMBDA[l] + 0.5 * rho * (tradeLin[l] + tradeLin[k]);
		Bt1[l] = 0.5 * (tradeLin[l] - tradeLin[k]) - m / rho;
		LAMBDA[l] = m;
	}
}


__global__ void updateDiffGPU(float* tempN, float* Tlocal, float* CoresLinTrans, int const N)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int step = blockDim.x * gridDim.x;
	for (int l = index; l < N; l += step)
	{
		int k = CoresLinTrans[l];
		tempN[l] = (Tlocal[l] + Tlocal[k]);
	}
}

__global__ void updateResKappa(float* result, float* Kappa1, float* Kappa2, float* Kappapre1, float* Kappapre2 ,float ratio, int const L)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int step = blockDim.x * gridDim.x;
	for (int l = index; l < L; l += step)
	{
		float kappaNeg1 = Kappa1[l] < 0 ? Kappa1[l] : 0;
		float kappaNeg2 = Kappa2[l] < 0 ? Kappa2[l] : 0;
		float kappaNegpre1 = Kappapre1[l] < 0 ? Kappapre1[l] : 0;
		float kappaNegpre2 = Kappapre2[l] < 0 ? Kappapre2[l] : 0;

		float res1 = kappaNeg1 - kappaNegpre1;
		res1 *= res1;
		float res2 = kappaNeg2 - kappaNegpre2;
		res2 *= res2;

		result[l] = ratio * sqrt(res1 + res2);
	}
}


__global__ void selectResidual(float* res, unsigned int id1, unsigned int id2, unsigned int id3, float* output) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index == 0) {
		float max = res[id1] > res[id2] ? res[id1] : res[id2];
		max = res[id3] > max ? res[id3] : max;
		*output = max;
	}
}

__global__ void updateKappaGPU(float* Kappa1, float* Kappa2, float* Llimit, float* Qtot, int nLine)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int step = blockDim.x * gridDim.x;
	for (int l = index; l < nLine; l += step)
	{
		float kappaNeg1 = Kappa1[l] < 0 ? Kappa1[l] : 0;
		float kappaNeg2 = Kappa2[l] < 0 ? Kappa2[l] : 0;
		float lim = Llimit[l];
		float Q = Qtot[l];
		Kappa1[l] = kappaNeg1 + lim - Q;
		Kappa2[l] = kappaNeg2 + lim + Q;
	}
}
__global__ void diffKappa(float* tempL1, float* Kappa1, float* Kappa2, int nLine)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int step = blockDim.x * gridDim.x;
	for (int l = index; l < nLine; l += step)
	{
		float Kappa1Abs = Kappa1[l] > 0 ? Kappa1[l] : -Kappa1[l]; //2 * (Kappa1[l] > 0) * Kappa1[l] - Kappa1[l]; // Kappa1[l] > 0 ? Kappa1[l] : -Kappa1[l]
		float Kappa2Abs = Kappa2[l] > 0 ? Kappa2[l] : -Kappa2[l]; //2 * (Kappa2[l] > 0) * Kappa2[l] - Kappa2[l]; // Kappa2[l] > 0 ? Kappa2[l] : -Kappa2[l]
		tempL1[l] = Kappa1Abs - Kappa2Abs;
	}
}


__global__ void updateCpOld(float* Cp, float* Cp1, float* Cp2, float* tempN1, float* nVoisin, const float rho1, const int nAgent) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int step = blockDim.x * gridDim.x;
	for (int n = index; n < nAgent; n += step)
	{
		Cp[n] = Cp1[n] + rho1 * (nVoisin[n] * (Cp2[n] + tempN1[n]));
	}
}

__global__ void updateCp(float* Cp, float* Cp1, float* Cp2, const int nAgent) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int step = blockDim.x * gridDim.x;
	for (int n = index; n < nAgent; n += step)
	{
		Cp[n] = Cp1[n] + Cp2[n];
	}
}

__global__ void updateQpart(float* Qpart, float* alpha, const int nAgent) {
	int index = threadIdx.x;
	int step = blockDim.x;
	int l = blockIdx.x;
	extern __shared__ float shAlpha[];

	for (int n = index; n < nAgent; n += step)
	{
		shAlpha[n] = alpha[l * nAgent + n];
	}
	__syncthreads();

	float s_pre = 0;
	int n_pre = nAgent - 1;
	for (int n = (nAgent - index - 1); n >= 0; n -= step)
	{
		float s = 0;
		for (int j = n_pre; j > n; j--) {
			s += shAlpha[j]; // c'est moche cet acc�s de m�moire partag�e
		}
		s = s + s_pre;
		Qpart[l * nAgent + n] = s;
		s_pre = s;
		n_pre = n;
	}
}
__global__ void updateQpartTrans(float* Qpart, float* alpha, const int N, const int nLine) {
	int index = threadIdx.x;
	int step = blockDim.x;
	int l = blockIdx.x;
	extern __shared__ float shAlpha[];

	for (int n = index; n < N; n += step)
	{
		shAlpha[n] = alpha[n * nLine + l]; // moche
	}
	__syncthreads();
	float s_pre = 0;
	int n_pre = N - 1;
	for (int n = (N - index - 1); n >= 0; n -= step)
	{
		float s = 0;
		for (int j = n_pre; j > n; j--) {
			s += shAlpha[j]; // c'est moche cet acc�s de m�moire partag�e
		}
		s = s + s_pre;
		Qpart[n * nLine + l] = s;
		s_pre = s;
		n_pre = n;
	}
}


__global__ void updateQtot(float* Qtot, float* Qpart, float* alpha, const int nLine, const int nAgent) {


	int thIdx = threadIdx.x + blockIdx.x * blockDim.x;
	int step = blockDim.x * gridDim.x;

	for (int l = thIdx; l < nLine; l += step) {

		Qtot[l] = Qpart[l * nAgent] + alpha[l * nAgent];
	}
}
__global__ void updateQtotTrans(float* Qtot, float* Qpart, float* alpha, const int nLine) {


	int thIdx = threadIdx.x + blockIdx.x * blockDim.x;
	int step = blockDim.x * gridDim.x;

	for (int l = thIdx; l < nLine; l += step) {

		Qtot[l] = Qpart[l] + alpha[l];
	}
}

__global__ void updateAlpha(float* alpha, float* G, float* Pn, const int nLine, const int nAgent)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	int step = blockDim.x * gridDim.x;
	int N = nAgent * nLine;

	for (int i = index; i < N; i += step)
	{
		int k = i % nAgent;
		alpha[i] = G[i] * Pn[k];
	}
}
__global__ void updateAlphaTrans(float* alpha, float* GTrans, float* Pn, const int nLine, const int nAgent) {

	int index = threadIdx.x + blockIdx.x * blockDim.x;
	int step = blockDim.x * gridDim.x;
	int N = nAgent * nLine;

	for (int i = index; i < N; i += step)
	{
		int k = i / nLine;
		alpha[i] = GTrans[i] * Pn[k];
	}

}


__global__ void updateResX(float* res, float* Kappa1, float* Kappa2, float* KappaPre1, float* KappaPre2, const int nLine) {

	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int step = blockDim.x * gridDim.x;
	for (int l = index; l < nLine; l += step)
	{
		float k1 = (Kappa1[l] < 0) * Kappa1[l];
		float k2 = (Kappa2[l] < 0) * Kappa2[l];
		float k1pre = (KappaPre1[l] < 0) * KappaPre1[l];
		float k2pre = (KappaPre2[l] < 0) * KappaPre2[l];

		k1 -= k1pre;
		k2 -= k2pre;

		res[l] = k1 * k1 + k2 * k2;
	}
}



__global__ void updatePnGPU(float* Pn, float* Tmoy, float* nVoisin, const int nAgent)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int step = blockDim.x * gridDim.x;
	for (int n = index; n < nAgent; n += step)
	{
		Pn[n] = Tmoy[n] * nVoisin[n];
	}

}



/*
__global__ void updateUAiq(float* UAiq, float* u, float* Aiq, int N, int size) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int step = blockDim.x * gridDim.x;
	for (int n = index; n < size; n += step)
	{
		int row = n / N;
		UAiq[n] = u[row] * Aiq[n];
	}
}

// Ru = U*g + epsi
__global__ void updateRu(float* Ru, float* U, float* g, float epsi, int N, int L2) {

	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int step = blockDim.x * gridDim.x;
	for (int i = index; i < L2; i += step)
	{
		
		Ru[i + N] = U[i] * g[i] + epsi;
	}

}

__global__ void updateV(float* v, float* pas, float* alpha, int offset) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index == 0) {
		*v = *v + (*alpha) * pas[offset];
	}
}

__global__ void updateQt(float* qt, float* Pso, float* Pn, float* etaSO, float rho1, int N) {

	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int step = blockDim.x * gridDim.x;
	for (int i = index; i < N; i += step)
	{

		qt[i] = etaSO[i] - rho1 * (Pso[i] + Pn[i]) / 2;
	}


}*/

__global__ void updatePI(float* PI, float* c, float mu, float valMin, int L) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int step = blockDim.x * gridDim.x;
	for (int i = index; i < L; i += step)
	{
		PI[i] = c[i] < valMin ? mu / valMin : mu / c[i];
	}
	if (index == 0) {
		PI[L] = -c[L] / mu;
	}
}


__global__ void updatePso(float* Pso, float* pas, float* alpha, int N) {

	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int step = blockDim.x * gridDim.x;
	for (int i = index; i < N; i += step)
	{

		Pso[i] = Pso[i] + *alpha * pas[i];
	}

}

__global__ void updateU(float* U, float* pas, float* alpha, int N, int L2) {

	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int step = blockDim.x * gridDim.x;
	for (int i = index; i < L2; i += step)
	{

		U[i] = U[i] + (*alpha) * pas[i + N];
	}

}


__global__ void updateEtaPBp3(float* Bp3, float* etaP, float* nVoisin, float* Pso, float* Pn, float rho, const int nAgent) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int step = blockDim.x * gridDim.x;
	for (int l = index; l < nAgent; l += step)
	{
		
		float m = etaP[l] + 0.5 * rho * (Pso[l] - Pn[l]);
		Bp3[l] = (0.5 * (Pso[l] + Pn[l]) + m / rho) / nVoisin[l];
		etaP[l] = m;
	}


}