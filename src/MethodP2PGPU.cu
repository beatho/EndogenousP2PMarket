#include "hip/hip_runtime.h"
#include "../head/MethodP2PGPU.cuh"
 


MethodP2PGPU::MethodP2PGPU(){

}
MethodP2PGPU::~MethodP2PGPU(){

}


float MethodP2PGPU::updateRes(int iter)
{
	float resS = Tlocal.max2(&tradeLin);
	updateDiffGPU << <_numBlocksM, _blockSize >> > (tempNN._matrixGPU, tradeLin._matrixGPU, CoresLinTrans._matrixGPU, _nTrade);
	float resR = tempNN.max2();

	updateResX << <_numBlocksL, _blockSize >> > (tempL1._matrixGPU, Kappa1._matrixGPU, Kappa2._matrixGPU, Kappa1_pre._matrixGPU, Kappa2_pre._matrixGPU, _nLine);

	resF.set(0, iter, resR);
	resF.set(1, iter, resS);
	if (iter > 0 && _tau > 1) {
		if (resR > _mu * resS) {
			_rhog = _tau * _rhog;
			_at1 = _rhog;
			//std::cout << iter << ", rho augmente :" << _rhog << std::endl;
		}
		else if (resS > _mu * resR) {// rho = rho / tau_inc;
			_rhog = _rhog / _tau;
			_at1 = _rhog;
			//std::cout << iter << ", rho diminue :" << _rhog << std::endl;
		}
	}
	
	return MYMAX(resS, resR);
}

float MethodP2PGPU::updateResEndo(int iter)
{
	float resS = Tlocal.max2(&tradeLin);
	updateDiffGPU << <_numBlocksM, _blockSize >> > (tempNN._matrixGPU, tradeLin._matrixGPU, CoresLinTrans._matrixGPU, _nTrade);
	float resR = tempNN.max2();

	updateResX << <_numBlocksL, _blockSize >> > (tempL1._matrixGPU, Kappa1._matrixGPU, Kappa2._matrixGPU, Kappa1_pre._matrixGPU, Kappa2_pre._matrixGPU, _nLine);


	float resXf = _ratioEps * sqrt(tempL1.max2());
	

	resF.set(0, iter, resR);
	resF.set(1, iter, resS);
	resF.set(2, iter, resXf);
	
	return MYMAX(MYMAX(resXf, resS), resR);
}

float MethodP2PGPU::calcRes()
{
	float d1 = Tlocal.max2(&Tlocal_pre);
	float d2 = P.max2(&Tmoy);

	return d1* (d1 > d2) + d2 * (d2 >= d1);
}


void MethodP2PGPU::updateLAMBDA(MatrixGPU* LAMBDA, MatrixGPU* trade, float rho, MatrixGPU* tempNN)
{
	tempNN->set(trade);
	tempNN->addTrans(trade);
	tempNN->multiply(rho);
	tempNN->multiply(0.5);
	LAMBDA->add(LAMBDA, tempNN);
}

void MethodP2PGPU::updateKappa()
{
	Kappa1.projectNeg();
	Kappa1.add(&lLimit);
	Kappa1.subtract(&Qtot);
	Kappa2.projectNeg();
	Kappa2.add(&lLimit);
	Kappa2.add(&Qtot);
}


void MethodP2PGPU::updatePn()
{
	Pn.set(&Tmoy);
	Pn.multiplyT(&nVoisin);
}


void MethodP2PGPU::solveWithMinPower(Simparam* result, const Simparam& sim, const StudyCase& cas)
{
	std::cout << "solveWithMinPower : should not be called" << std::endl;
}




float MethodP2PGPU::calcFc()
{
	
	tempN1.set(&a);
	
	tempN1.multiply(0.5);
	tempN1.multiplyT(&Pn);
	
	tempN1.add(&b);
	
	tempN1.multiplyT(&Pn);
	
	float fc = tempN1.sum();
	

	tempNN.set(&trade);
	
	tempNN.multiplyT(&Ct);
	
	fc = fc + tempNN.sum();



	//std::cout << "fc " << fc << std::endl;
	return fc;

}


void MethodP2PGPU::display(){
	std::cout << " resolution par la methode " << _name << std::endl;
}



__global__ void updateLAMBDAGPU(float* LAMBDALin, float* tradeLin, float rho, float* CoresLinTrans, int const N)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int step = blockDim.x * gridDim.x;
	for (int l = index; l < N; l += step)
	{
		float m = LAMBDALin[l];
		int k = CoresLinTrans[l];
		LAMBDALin[l] = m + 0.5 * rho * (tradeLin[l] + tradeLin[k]);
	}
}
__global__ void updateBt1GPU(float* Bt1, float* tradeLin, float rho, float* LAMBDA, float* CoresLinTrans, int const N)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int step = blockDim.x * gridDim.x;
	for (int l = index; l < N; l += step)
	{
		int k = CoresLinTrans[l];
		Bt1[l] = 0.5 * (tradeLin[l] - tradeLin[k]) - LAMBDA[l] / rho;
	}

}

__global__ void updateLAMBDABt1GPU(float* Bt1, float* LAMBDA, float* tradeLin, float rho, float* CoresLinTrans, int const N) {

	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int step = blockDim.x * gridDim.x;
	for (int l = index; l < N; l += step)
	{
		int k = CoresLinTrans[l];
		float m = LAMBDA[l] + 0.5 * rho * (tradeLin[l] + tradeLin[k]);
		Bt1[l] = 0.5 * (tradeLin[l] - tradeLin[k]) - m / rho;
		LAMBDA[l] = m;
	}
}


__global__ void updateDiffGPU(float* tempN, float* Tlocal, float* CoresLinTrans, int const N)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int step = blockDim.x * gridDim.x;
	for (int l = index; l < N; l += step)
	{
		int k = CoresLinTrans[l];
		tempN[l] = (Tlocal[l] + Tlocal[k]);
	}
}

__global__ void updateResKappa(float* result, float* Kappa1, float* Kappa2, float* Kappapre1, float* Kappapre2 ,float ratio, int const L)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int step = blockDim.x * gridDim.x;
	for (int l = index; l < L; l += step)
	{
		float kappaNeg1 = Kappa1[l] < 0 ? Kappa1[l] : 0;
		float kappaNeg2 = Kappa2[l] < 0 ? Kappa2[l] : 0;
		float kappaNegpre1 = Kappapre1[l] < 0 ? Kappapre1[l] : 0;
		float kappaNegpre2 = Kappapre2[l] < 0 ? Kappapre2[l] : 0;

		float res1 = kappaNeg1 - kappaNegpre1;
		res1 *= res1;
		float res2 = kappaNeg2 - kappaNegpre2;
		res2 *= res2;

		result[l] = ratio * sqrt(res1 + res2);
	}
}


__global__ void selectResidual(float* res, unsigned int id1, unsigned int id2, unsigned int id3, float* output) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index == 0) {
		float max = res[id1] > res[id2] ? res[id1] : res[id2];
		max = res[id3] > max ? res[id3] : max;
		*output = max;
	}
}

__global__ void updateKappaGPU(float* Kappa1, float* Kappa2, float* Llimit, float* Qtot, int nLine)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int step = blockDim.x * gridDim.x;
	for (int l = index; l < nLine; l += step)
	{
		float kappaNeg1 = Kappa1[l] < 0 ? Kappa1[l] : 0;
		float kappaNeg2 = Kappa2[l] < 0 ? Kappa2[l] : 0;
		float lim = Llimit[l];
		float Q = Qtot[l];
		Kappa1[l] = kappaNeg1 + lim - Q;
		Kappa2[l] = kappaNeg2 + lim + Q;
	}
}
__global__ void diffKappa(float* tempL1, float* Kappa1, float* Kappa2, int nLine)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int step = blockDim.x * gridDim.x;
	for (int l = index; l < nLine; l += step)
	{
		float Kappa1Abs = Kappa1[l] > 0 ? Kappa1[l] : -Kappa1[l]; //2 * (Kappa1[l] > 0) * Kappa1[l] - Kappa1[l]; // Kappa1[l] > 0 ? Kappa1[l] : -Kappa1[l]
		float Kappa2Abs = Kappa2[l] > 0 ? Kappa2[l] : -Kappa2[l]; //2 * (Kappa2[l] > 0) * Kappa2[l] - Kappa2[l]; // Kappa2[l] > 0 ? Kappa2[l] : -Kappa2[l]
		tempL1[l] = Kappa1Abs - Kappa2Abs;
	}
}


__global__ void updateCpOld(float* Cp, float* Cp1, float* Cp2, float* tempN1, float* nVoisin, const float rho1, const int nAgent) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int step = blockDim.x * gridDim.x;
	for (int n = index; n < nAgent; n += step)
	{
		Cp[n] = Cp1[n] + rho1 * (nVoisin[n] * (Cp2[n] + tempN1[n]));
	}
}

__global__ void updateCp(float* Cp, float* Cp1, float* Cp2, const int nAgent) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int step = blockDim.x * gridDim.x;
	for (int n = index; n < nAgent; n += step)
	{
		Cp[n] = Cp1[n] + Cp2[n];
	}
}

__global__ void updateQpart(float* Qpart, float* alpha, const int nAgent) {
	int index = threadIdx.x;
	int step = blockDim.x;
	int l = blockIdx.x;
	extern __shared__ float shAlpha[];

	for (int n = index; n < nAgent; n += step)
	{
		shAlpha[n] = alpha[l * nAgent + n];
	}
	__syncthreads();

	float s_pre = 0;
	int n_pre = nAgent - 1;
	for (int n = (nAgent - index - 1); n >= 0; n -= step)
	{
		float s = 0;
		for (int j = n_pre; j > n; j--) {
			s += shAlpha[j]; // c'est moche cet acc�s de m�moire partag�e
		}
		s = s + s_pre;
		Qpart[l * nAgent + n] = s;
		s_pre = s;
		n_pre = n;
	}
}
__global__ void updateQpartTrans(float* Qpart, float* alpha, const int N, const int nLine) {
	int index = threadIdx.x;
	int step = blockDim.x;
	int l = blockIdx.x;
	extern __shared__ float shAlpha[];

	for (int n = index; n < N; n += step)
	{
		shAlpha[n] = alpha[n * nLine + l]; // moche
	}
	__syncthreads();
	float s_pre = 0;
	int n_pre = N - 1;
	for (int n = (N - index - 1); n >= 0; n -= step)
	{
		float s = 0;
		for (int j = n_pre; j > n; j--) {
			s += shAlpha[j]; // c'est moche cet acc�s de m�moire partag�e
		}
		s = s + s_pre;
		Qpart[n * nLine + l] = s;
		s_pre = s;
		n_pre = n;
	}
}


__global__ void updateQtot(float* Qtot, float* Qpart, float* alpha, const int nLine, const int nAgent) {


	int thIdx = threadIdx.x + blockIdx.x * blockDim.x;
	int step = blockDim.x * gridDim.x;

	for (int l = thIdx; l < nLine; l += step) {

		Qtot[l] = Qpart[l * nAgent] + alpha[l * nAgent];
	}
}
__global__ void updateQtotTrans(float* Qtot, float* Qpart, float* alpha, const int nLine) {


	int thIdx = threadIdx.x + blockIdx.x * blockDim.x;
	int step = blockDim.x * gridDim.x;

	for (int l = thIdx; l < nLine; l += step) {

		Qtot[l] = Qpart[l] + alpha[l];
	}
}

__global__ void updateAlpha(float* alpha, float* G, float* Pn, const int nLine, const int nAgent)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	int step = blockDim.x * gridDim.x;
	int N = nAgent * nLine;

	for (int i = index; i < N; i += step)
	{
		int k = i % nAgent;
		alpha[i] = G[i] * Pn[k];
	}
}
__global__ void updateAlphaTrans(float* alpha, float* GTrans, float* Pn, const int nLine, const int nAgent) {

	int index = threadIdx.x + blockIdx.x * blockDim.x;
	int step = blockDim.x * gridDim.x;
	int N = nAgent * nLine;

	for (int i = index; i < N; i += step)
	{
		int k = i / nLine;
		alpha[i] = GTrans[i] * Pn[k];
	}

}


__global__ void updateResX(float* res, float* Kappa1, float* Kappa2, float* KappaPre1, float* KappaPre2, const int nLine) {

	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int step = blockDim.x * gridDim.x;
	for (int l = index; l < nLine; l += step)
	{
		float k1 = (Kappa1[l] < 0) * Kappa1[l];
		float k2 = (Kappa2[l] < 0) * Kappa2[l];
		float k1pre = (KappaPre1[l] < 0) * KappaPre1[l];
		float k2pre = (KappaPre2[l] < 0) * KappaPre2[l];

		k1 -= k1pre;
		k2 -= k2pre;

		res[l] = k1 * k1 + k2 * k2;
	}
}



__global__ void updatePnGPU(float* Pn, float* Tmoy, float* nVoisin, const int nAgent)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int step = blockDim.x * gridDim.x;
	for (int n = index; n < nAgent; n += step)
	{
		Pn[n] = Tmoy[n] * nVoisin[n];
	}

}



/*
__global__ void updateUAiq(float* UAiq, float* u, float* Aiq, int N, int size) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int step = blockDim.x * gridDim.x;
	for (int n = index; n < size; n += step)
	{
		int row = n / N;
		UAiq[n] = u[row] * Aiq[n];
	}
}

// Ru = U*g + epsi
__global__ void updateRu(float* Ru, float* U, float* g, float epsi, int N, int L2) {

	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int step = blockDim.x * gridDim.x;
	for (int i = index; i < L2; i += step)
	{
		
		Ru[i + N] = U[i] * g[i] + epsi;
	}

}

__global__ void updateV(float* v, float* pas, float* alpha, int offset) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index == 0) {
		*v = *v + (*alpha) * pas[offset];
	}
}

__global__ void updateQt(float* qt, float* Pso, float* Pn, float* etaSO, float rho1, int N) {

	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int step = blockDim.x * gridDim.x;
	for (int i = index; i < N; i += step)
	{

		qt[i] = etaSO[i] - rho1 * (Pso[i] + Pn[i]) / 2;
	}


}*/

__global__ void updatePI(float* PI, float* c, float mu, float valMin, int L) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int step = blockDim.x * gridDim.x;
	for (int i = index; i < L; i += step)
	{
		PI[i] = c[i] < valMin ? mu / valMin : mu / c[i];
	}
	if (index == 0) {
		PI[L] = -c[L] / mu;
	}
}


__global__ void updatePso(float* Pso, float* pas, float* alpha, int N) {

	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int step = blockDim.x * gridDim.x;
	for (int i = index; i < N; i += step)
	{

		Pso[i] = Pso[i] + *alpha * pas[i];
	}

}

__global__ void updateU(float* U, float* pas, float* alpha, int N, int L2) {

	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int step = blockDim.x * gridDim.x;
	for (int i = index; i < L2; i += step)
	{

		U[i] = U[i] + (*alpha) * pas[i + N];
	}

}


__global__ void updateEtaPBp3(float* Bp3, float* etaP, float* nVoisin, float* Pso, float* Pn, float rho, const int nAgent) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int step = blockDim.x * gridDim.x;
	for (int l = index; l < nAgent; l += step)
	{
		
		float m = etaP[l] + 0.5 * rho * (Pso[l] - Pn[l]);
		Bp3[l] = (0.5 * (Pso[l] + Pn[l]) + m / rho) / nVoisin[l];
		etaP[l] = m;
	}


}