#include "hip/hip_runtime.h"
#include "../head/ADMMGPUConstCons3.cuh"

ADMMGPUConstCons3::ADMMGPUConstCons3() : MethodP2PGPU()
{
#if DEBUG_CONSTRUCTOR
	std::cout << "Constructeur ADMMGPUConstCons3" << std::endl;
#endif // DEBUG_CONSTRUCTOR
	_name = NAME;
	timePerBlock = MatrixCPU(1, 9, 0); // Fb0, Fb1 , Fb2, Fb3, Fb5, Fb6 Fb0'
	// si les sous ensemble ne sont pas accessible, tout est dans le premier.
	occurencePerBlock = MatrixCPU(1, 9, 0); //nb de fois utilis� pendant la simu

}


ADMMGPUConstCons3::ADMMGPUConstCons3(float rho) : MethodP2PGPU()
{
#if DEBUG_CONSTRUCTOR
	std::cout << "Constructeur ADMMGPUConstCons3 defaut" << std::endl;
#endif // DEBUG_CONSTRUCTOR
	_name = NAME;
	_rho = rho;
	timePerBlock = MatrixCPU(1, 9, 0); // Fb0, Fb1 , Fb2, Fb3, Fb5, Fb6 Fb0'
	// si les sous ensemble ne sont pas accessible, tout est dans le premier.
	occurencePerBlock = MatrixCPU(1, 9, 0); //nb de fois utilis� pendant la simu

}

ADMMGPUConstCons3::~ADMMGPUConstCons3()
{
	if (alpha != nullptr) {
		//std::cout << "delete alpha" << std::endl;
		hipFree(alpha);
		alpha = nullptr;
	}
}

void ADMMGPUConstCons3::setParam(float rho)
{
	_rho = rho;
}

void ADMMGPUConstCons3::setTau(float tau)
{
	if (tau < 1) {
		throw std::invalid_argument("tau must be greater than 1");
	}
	_tau = tau;
}

void ADMMGPUConstCons3::init(const Simparam& sim, const StudyCase& cas)
{
	// intitilisation des matrixs et variables 
	isAC = false;
	initSize(cas);
	initSimParam(sim);

	int nVoisinMax = nVoisin.max2();
	if (_blockSize * NMAXPEERPERTRHREAD < nVoisinMax) {
		std::cout << _blockSize << " " << NMAXPEERPERTRHREAD << " " << nVoisinMax << std::endl;
		throw std::invalid_argument("For this Method, an agent must not have more than 5120 peers");
	}

	
	L2 = 2 * _nLine;
	_Msize = _nAgent + L2 + 1;
	_Asize = L2 * _nAgent;
	//std::cout << _nAgent << " " << _nLine << " " << _Msize << std::endl;
	initCaseParam(sim, cas);
	
	//std::cout << "mise sous forme lineaire" << std::endl;
	initLinForm(cas);
		
	//std::cout << "donnees sur GPU pour le grid" << std::endl;
	if (_nLine) {
		hipMalloc((void**)&alpha, sizeof(float));
		tempL21 = MatrixGPU(L2 + 1, 1, 0, 1);

		H = MatrixGPU(_nAgent, _nAgent, 0, 1);
		H.setEyes(_rho1);

		q = MatrixGPU(_nAgent, 1, 0, 1); // 0.5x^THx + q^T*x
		diffPso = MatrixGPU(_nAgent, 1, 0, 1);

		c = MatrixGPU(L2 + 1, 1, 0, 1); // contrainte Ax+b>0 ou = 0 pour egalit�
		Ai = MatrixGPU(L2 + 1, _nAgent, 0, 1);
		MatrixGPU ones(1, _nAgent, 1, 1);
		MatrixGPU temp(cas.getPowerSensi(), 1);
		Ai.setBloc(0, _nLine, 0, _nAgent, &temp, -1);
		Ai.setBloc(_nLine, L2, 0, _nAgent, &temp);
		Ai.setBloc(L2, L2 + 1, 0, _nAgent, &ones);
		bi = MatrixGPU(L2 + 1, 1, 0, 1);
		lLimit = MatrixGPU(cas.getLineLimit(), 1);
		bi.setBloc(0, _nLine, 0, 1, &lLimit);
		bi.setBloc(_nLine, L2, 0, 1, &lLimit);

		M = MatrixGPU(_Msize, _Msize, 0, 1); // M*pas = R
		Minv = MatrixGPU(_Msize, _Msize, 0, 1); // M*pas = R
		pas = MatrixGPU(_Msize, 1, 0, 1);
		R = MatrixGPU(_Msize, 1, 0, 1);


		ZA = MatrixGPU(L2 + 1, _nAgent, 0, 1); // M = (H -Atrans ZA W)
		Z = MatrixGPU(L2 + 1, L2 + 1, 0, 1);
		Zvect = MatrixGPU(L2 + 1, 1, 0, 1);
		W = MatrixGPU(L2 + 1, L2 + 1, 0, 1);
		Wvect = MatrixGPU(L2 + 1, 1, 0, 1);
		Atrans = MatrixGPU(_nAgent, L2 + 1, 0, 1);
		Atrans.setTrans(&Ai);

		M.setBloc(0, _nAgent, 0, _nAgent, &H);
		M.setBloc(0, _nAgent, _nAgent, _Msize, &Atrans, -1);

		Rx1 = MatrixGPU(_nAgent, 1, 0, 1); // Hx+q
		Rx2 = MatrixGPU(_nAgent, 1, 0, 1); // -Ai^T*U

		Ru = MatrixGPU(L2 + 1, 1, 0, 1); // Ru = W*(U-PI)

		U = MatrixGPU(L2 + 1, 1, 0, 1);
		PI = MatrixGPU(L2 + 1, 1, 0, 1);
		
		Apas = MatrixGPU(L2 + 1, 1, 0, 1);
	}
	Pso = MatrixGPU(_nAgent, 1, 0, 1); // = Pn ? risque de non respect des contraintes
	etaP = MatrixGPU(_nAgent, 1, 0, 1); 
		
	R.preallocateReduction();
	Pso.preallocateReduction();

	//std::cout << "autres donn�e sur GPU" << std::endl;
	
	initDCEndoMarket();

	Ap3 = nVoisin;
	Bp3 = MatrixGPU(_nAgent, 1, 0, 1); // 1/Mn * (Pso + P)/2 - eta/rho1
	Ap3.multiplyT(&nVoisin);
	Ap3.multiply(_rho1);
	Ap123.add(&Ap12, &Ap3);
	
	
	updateGlobalProbGPU();

	//std::cout << " end init " << std::endl;
}

void ADMMGPUConstCons3::solveOPF()
{
	// update q
    diffPso.set(&Pso);
	q.set(&etaP);
	diffPso.add(&Pn);
	diffPso.multiply(-_rho1/2);
	q.add(&diffPso);
	
	//init
	int k = 0;
		
	float err = 2 * _epsIntern;
	mu = 10;
	float valMin = 0.0000001;
	//boucle
	while (k< _iterIntern && err > _epsIntern) {
	// update c
		
		c.linearOperation(&Ai, &Pso, &bi);
		
	// update PI
		/*c.transferCPU();
		PI.transferCPU();
		for (int l = 0; l < L2; l++) {
			if (c.get(l, 0) < valMin) {
				PI.set(l, 0, mu / valMin); // eviter division par O
			}
			else {
				PI.set(l, 0, mu / c.get(l, 0)); // eviter division par O
			}
		}
		PI.set(L2, 0, -c.get(L2, 0) / mu);
		PI.transferGPU();
		c.transferGPU();*/
		updatePI << <_numBlocksL, _blockSize >> > (PI._matrixGPU, c._matrixGPU, mu, valMin, L2);
	
	// update M
		// update Zvect
		Zvect.set(&U); 
		
		Zvect.set(L2, 0, 1, true); // egalite
		// update Z
		Z.setEyes(&Zvect);
		// update ZA
		ZA.multiplyMat(&Z, &Ai);
		// update W
		Wvect.set(&c);
		Wvect.set(L2, 0, mu, true);
		W.setEyes(&Wvect);

		M.setBloc(_nAgent, _Msize, 0, _nAgent, &ZA);
		M.setBloc(_nAgent, _Msize, _nAgent, _Msize, &W);
		try
		{
			Minv.invertGaussJordan(&M);
		}
		catch (const std::exception& e)
		{
			std::cout << e.what() << std::endl;
			float alphaCPU;
			hipMemcpy(&alphaCPU, alpha, sizeof(float), hipMemcpyDeviceToHost);
			std::cout << "k = " << k << " err= " << err << " alpha = " << alphaCPU  << " mu=" << mu << std::endl;
			c.display(true);
			Pn.display(true);
			Pso.display(true);
			std::cout << "---------------------------------" << std::endl;
			Pso.set(&Pn);
			return;
		}
		
	
	//update R
		// Rx
		Rx1.linearOperation(&H, &Pso, &q);
		Rx2.multiply(&Atrans, &U);
		Rx2.subtract(&Rx1);
		// Ru
		tempL21.subtract(&PI, &U);
		Ru.multiply(&W, &tempL21);

		R.setBloc(0, _nAgent, 0, 1, &Rx2);
		R.setBloc(_nAgent, _Msize, 0, 1, &Ru);
		//update pas
		pas.multiply(&Minv, &R);
		// find alpha
		findalpha();
		/*float alphaCPU;
		hipMemcpy(&alphaCPU, alpha, sizeof(float), hipMemcpyDeviceToHost);
		
		if (alphaCPU < valMin) { // trop proche de la froncti�re
			std::cout << "alpha " << alphaCPU << std::endl;
			c.display(true);
			Pso.display(true);
			Pn.display(true);
			break;
		}*/
		
		// update P, U
		updatePso << <_numBlocksN, _blockSize >> > (Pso._matrixGPU, pas._matrixGPU, alpha, _nAgent);
		updateU << <_numBlocksL, _blockSize >> > (U._matrixGPU, pas._matrixGPU, alpha, _nAgent, L2 + 1);

		

		// update mu
		mu *= 0.8;
		mu = MYMAX(mu, valMin);
		//R.transferCPU();
		err = R.distance2();
		//R.transferGPU();
		k++;
	}
	//std::cout << k << " " << err << std::endl;
}



void ADMMGPUConstCons3::findalpha()
{
	

	int numBlock = L2;
	switch (_blockSize) {
	case 512:
		updateAPas<512> << <numBlock, _blockSize >> > (Apas._matrixGPU, Ai._matrixGPU, pas._matrixGPU, _nAgent);
		updateAlpha<512> << <1, _blockSize >> > (alpha, U._matrixGPU, pas._matrixGPU, c._matrixGPU, Apas._matrixGPU, _nAgent, L2);
		break;
	case 256:
		updateAPas<256> << <numBlock, _blockSize >> > (Apas._matrixGPU, Ai._matrixGPU, pas._matrixGPU, _nAgent);
		updateAlpha<256> << <1, _blockSize >> > (alpha, U._matrixGPU, pas._matrixGPU, c._matrixGPU, Apas._matrixGPU, _nAgent, L2);
		break;
	case 128:
		updateAPas<128> << <numBlock, _blockSize >> > (Apas._matrixGPU, Ai._matrixGPU, pas._matrixGPU, _nAgent);
		updateAlpha<128> << <1, _blockSize >> > (alpha, U._matrixGPU, pas._matrixGPU, c._matrixGPU, Apas._matrixGPU, _nAgent, L2);
		break;
	case 64:
		updateAPas< 64> << <numBlock, _blockSize >> > (Apas._matrixGPU, Ai._matrixGPU, pas._matrixGPU, _nAgent);
		updateAlpha< 64> << <1, _blockSize >> > (alpha, U._matrixGPU, pas._matrixGPU, c._matrixGPU, Apas._matrixGPU, _nAgent, L2);
		break;
	case 32:
		updateAPas< 32> << <numBlock, _blockSize >> > (Apas._matrixGPU, Ai._matrixGPU, pas._matrixGPU, _nAgent);
		updateAlpha< 32> << <1, _blockSize >> > (alpha, U._matrixGPU, pas._matrixGPU, c._matrixGPU, Apas._matrixGPU, _nAgent, L2);
		break;
	case 16:
		updateAPas< 16> << <numBlock, _blockSize >> > (Apas._matrixGPU, Ai._matrixGPU, pas._matrixGPU, _nAgent);
		updateAlpha< 16> << <1, _blockSize >> > (alpha, U._matrixGPU, pas._matrixGPU, c._matrixGPU, Apas._matrixGPU, _nAgent, L2);
		break;
	case  8:
		updateAPas<  8> << <numBlock, _blockSize >> > (Apas._matrixGPU, Ai._matrixGPU, pas._matrixGPU, _nAgent);
		updateAlpha< 8> << <1, _blockSize >> > (alpha, U._matrixGPU, pas._matrixGPU, c._matrixGPU, Apas._matrixGPU, _nAgent, L2);
		break;
	case  4:
		updateAPas<  4> << <numBlock, _blockSize >> > (Apas._matrixGPU, Ai._matrixGPU, pas._matrixGPU, _nAgent);
		updateAlpha<  4> << <1, _blockSize >> > (alpha, U._matrixGPU, pas._matrixGPU, c._matrixGPU, Apas._matrixGPU, _nAgent, L2);
		break;
	case  2:
		updateAPas<  2> << <numBlock, _blockSize >> > (Apas._matrixGPU, Ai._matrixGPU, pas._matrixGPU, _nAgent);
		updateAlpha<  2> << <1, _blockSize >> > (alpha, U._matrixGPU, pas._matrixGPU, c._matrixGPU, Apas._matrixGPU, _nAgent, L2);
		break;
	case  1:
		updateAPas<  1> << <numBlock, _blockSize >> > (Apas._matrixGPU, Ai._matrixGPU, pas._matrixGPU, _nAgent);
		updateAlpha<  1> << <1, _blockSize >> > (alpha, U._matrixGPU, pas._matrixGPU, c._matrixGPU, Apas._matrixGPU, _nAgent, L2);
		break;
	}
	

	// truc avec alpha et beta pour optimiser sa valeur


}




void ADMMGPUConstCons3::solve(Simparam* result, const Simparam& sim, const StudyCase& cas)
{
#ifdef DEBUG_SOLVE
	cas.display();
	sim.display(1);
#endif // DEBUG_SOLVE
	
	
	tMarket = clock();
#ifdef INSTRUMENTATION
	std::chrono::high_resolution_clock::time_point t1;
	std::chrono::high_resolution_clock::time_point t2;
#endif // INSTRUMENTATION

	if (_id == 0) {
#ifdef INSTRUMENTATION
		hipDeviceSynchronize();
		t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION
		init(sim, cas);
#ifdef INSTRUMENTATION
		hipDeviceSynchronize();
		t2 = std::chrono::high_resolution_clock::now();
		timePerBlock.increment(0, 0, (float) std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
		occurencePerBlock.increment(0, 0, 1);
#endif // INSTRUMENTATION
	}
	//std::cout << _numBlocks2 << " " <<  _blockSize << std::endl;
	_rhog = sim.getRho();
	_at1 = _rhog; // represente en fait 2*a
	
	
	float resG = 2 * _epsG;
	float epsL2 = _epsL * _epsL;
	_iterGlobal = 0;
	
	//std::cout << iterG << " " << iterL << " " << epsL << " " << epsG << std::endl;
	while ((_iterGlobal < _iterG) && (resG > _epsG)) {
#ifdef INSTRUMENTATION
		hipDeviceSynchronize();
		t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION

		updateLocalProbGPU(epsL2, _iterL);
#ifdef INSTRUMENTATION
		hipDeviceSynchronize();
		t2 = std::chrono::high_resolution_clock::now();
		timePerBlock.increment(0, 1, (float) std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
#endif // INSTRUMENTATION

		tradeLin.swap(&Tlocal); // echange juste les pointeurs	
		//std::cout << "-";
		
		updateGlobalProbGPU();
		
		if (!(_iterGlobal % _stepG)) {
#ifdef INSTRUMENTATION
			hipDeviceSynchronize();
			t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION
			resG = updateResEndo(_iterGlobal / _stepG);
#ifdef INSTRUMENTATION
			hipDeviceSynchronize();
			t2 = std::chrono::high_resolution_clock::now();
			timePerBlock.increment(0, 8, (float) std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
#endif // INSTRUMENTATION
		}
		//std::cout << iterGlobal << " " << iterLocal << " " << resL << " " << resF.get(0, iterGlobal / stepG) << " " << resF.get(1, iterGlobal / stepG) << std::endl;
		_iterGlobal++;
	}
#ifdef INSTRUMENTATION
	occurencePerBlock.increment(0, 1, iterGlobal);
	occurencePerBlock.increment(0, 5, iterGlobal);
	occurencePerBlock.increment(0, 6, iterGlobal);
	occurencePerBlock.increment(0, 7, iterGlobal);
	occurencePerBlock.increment(0, 8, iterGlobal / stepG);

	hipDeviceSynchronize();
	t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION
	
	

	//std::cout << iterGlobal << " " << iterLocal << " " << resL << " " << resG << std::endl;
	std::cout << "valeur finale des contraintes de l'opf : " << std::endl;
	c.display(true);
	
	setResult(result, cas.isAC());


#ifdef INSTRUMENTATION
	hipDeviceSynchronize();
	t2 = std::chrono::high_resolution_clock::now();
	timePerBlock.increment(0, 9, (float) std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
	occurencePerBlock.increment(0, 9, 1);
	result->setTimeBloc(&timePerBlock, &occurencePerBlock);
#endif // INSTRUMENTATION
}

void ADMMGPUConstCons3::updateLocalProbGPU(float epsL, int nIterL) {
	int numBlocks = _nAgent;
	switch (_blockSize) {
	case 512:
		updateTradePGPUSharedResidualCons<512> << <numBlocks, _blockSize >> > (Tlocal._matrixGPU, Tlocal_pre._matrixGPU, Tmoy._matrixGPU, P._matrixGPU, MU._matrixGPU, nVoisin._matrixGPU, _at1, _at2, Bt1._matrixGPU, Ct._matrixGPU,
			matLb._matrixGPU, matUb._matrixGPU, Ap1._matrixGPU, Ap3._matrixGPU, Ap123._matrixGPU, Bp3._matrixGPU, Cp._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, CoresAgentLin._matrixGPU, epsL, nIterL);
		break;
	case 256:
		updateTradePGPUSharedResidualCons<256> << <numBlocks, _blockSize >> > (Tlocal._matrixGPU, Tlocal_pre._matrixGPU, Tmoy._matrixGPU, P._matrixGPU, MU._matrixGPU, nVoisin._matrixGPU, _at1, _at2, Bt1._matrixGPU, Ct._matrixGPU,
			matLb._matrixGPU, matUb._matrixGPU, Ap1._matrixGPU, Ap3._matrixGPU, Ap123._matrixGPU, Bp3._matrixGPU, Cp._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, CoresAgentLin._matrixGPU, epsL, nIterL);
		break;
	case 128:
		updateTradePGPUSharedResidualCons<128> << <numBlocks, _blockSize >> > (Tlocal._matrixGPU, Tlocal_pre._matrixGPU, Tmoy._matrixGPU, P._matrixGPU, MU._matrixGPU, nVoisin._matrixGPU, _at1, _at2, Bt1._matrixGPU, Ct._matrixGPU,
			matLb._matrixGPU, matUb._matrixGPU, Ap1._matrixGPU, Ap3._matrixGPU, Ap123._matrixGPU, Bp3._matrixGPU, Cp._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, CoresAgentLin._matrixGPU, epsL, nIterL);
		break;
	case 64:
		updateTradePGPUSharedResidualCons< 64> << <numBlocks, _blockSize >> > (Tlocal._matrixGPU, Tlocal_pre._matrixGPU, Tmoy._matrixGPU, P._matrixGPU, MU._matrixGPU, nVoisin._matrixGPU, _at1, _at2, Bt1._matrixGPU, Ct._matrixGPU,
			matLb._matrixGPU, matUb._matrixGPU, Ap1._matrixGPU, Ap3._matrixGPU, Ap123._matrixGPU, Bp3._matrixGPU, Cp._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, CoresAgentLin._matrixGPU, epsL, nIterL);
		break;
	case 32:
		updateTradePGPUSharedResidualCons< 32> << <numBlocks, _blockSize >> > (Tlocal._matrixGPU, Tlocal_pre._matrixGPU, Tmoy._matrixGPU, P._matrixGPU, MU._matrixGPU, nVoisin._matrixGPU, _at1, _at2, Bt1._matrixGPU, Ct._matrixGPU,
			matLb._matrixGPU, matUb._matrixGPU, Ap1._matrixGPU, Ap3._matrixGPU, Ap123._matrixGPU, Bp3._matrixGPU, Cp._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, CoresAgentLin._matrixGPU, epsL, nIterL);
		break;
	case 16:
		updateTradePGPUSharedResidualCons< 16> << <numBlocks, _blockSize >> > (Tlocal._matrixGPU, Tlocal_pre._matrixGPU, Tmoy._matrixGPU, P._matrixGPU, MU._matrixGPU, nVoisin._matrixGPU, _at1, _at2, Bt1._matrixGPU, Ct._matrixGPU,
			matLb._matrixGPU, matUb._matrixGPU, Ap1._matrixGPU, Ap3._matrixGPU, Ap123._matrixGPU, Bp3._matrixGPU, Cp._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, CoresAgentLin._matrixGPU, epsL, nIterL);
		break;
	case  8:
		updateTradePGPUSharedResidualCons<  8> << <numBlocks, _blockSize >> > (Tlocal._matrixGPU, Tlocal_pre._matrixGPU, Tmoy._matrixGPU, P._matrixGPU, MU._matrixGPU, nVoisin._matrixGPU, _at1, _at2, Bt1._matrixGPU, Ct._matrixGPU,
			matLb._matrixGPU, matUb._matrixGPU, Ap1._matrixGPU, Ap3._matrixGPU, Ap123._matrixGPU, Bp3._matrixGPU, Cp._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, CoresAgentLin._matrixGPU, epsL, nIterL);
		break;
	case  4:
		updateTradePGPUSharedResidualCons<  4> << <numBlocks, _blockSize >> > (Tlocal._matrixGPU, Tlocal_pre._matrixGPU, Tmoy._matrixGPU, P._matrixGPU, MU._matrixGPU, nVoisin._matrixGPU, _at1, _at2, Bt1._matrixGPU, Ct._matrixGPU,
			matLb._matrixGPU, matUb._matrixGPU, Ap1._matrixGPU, Ap3._matrixGPU, Ap123._matrixGPU, Bp3._matrixGPU, Cp._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, CoresAgentLin._matrixGPU, epsL, nIterL);
		break;
	case  2:
		updateTradePGPUSharedResidualCons<  2> << <numBlocks, _blockSize >> > (Tlocal._matrixGPU, Tlocal_pre._matrixGPU, Tmoy._matrixGPU, P._matrixGPU, MU._matrixGPU, nVoisin._matrixGPU, _at1, _at2, Bt1._matrixGPU, Ct._matrixGPU,
			matLb._matrixGPU, matUb._matrixGPU, Ap1._matrixGPU, Ap3._matrixGPU, Ap123._matrixGPU, Bp3._matrixGPU, Cp._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, CoresAgentLin._matrixGPU, epsL, nIterL);
		break;
	case  1:
		updateTradePGPUSharedResidualCons<  1> << <numBlocks, _blockSize >> > (Tlocal._matrixGPU, Tlocal_pre._matrixGPU, Tmoy._matrixGPU, P._matrixGPU, MU._matrixGPU, nVoisin._matrixGPU, _at1, _at2, Bt1._matrixGPU, Ct._matrixGPU,
			matLb._matrixGPU, matUb._matrixGPU, Ap1._matrixGPU, Ap3._matrixGPU, Ap123._matrixGPU, Bp3._matrixGPU, Cp._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, CoresAgentLin._matrixGPU, epsL, nIterL);
		break;
	}
	//hipStreamSynchronize(streamCalculation);
}



void ADMMGPUConstCons3::updateGlobalProbGPU()
{
	//Rem : tout calcul qui est de taille N ou M peut �tre fait par les agents
		// Si le calcul est de taile L, soit c'est calcul� par un/des superviseurs, soit tous les agents le calcul (un peu absurde)

#ifdef INSTRUMENTATION
// FB 3a
	hipDeviceSynchronize();
	std::chrono::high_resolution_clock::time_point t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION


	updatePnGPU << <_numBlocksN, _blockSize >> > (Pn._matrixGPU, Tmoy._matrixGPU, nVoisin._matrixGPU, _nAgent);
	
#ifdef INSTRUMENTATION
	hipDeviceSynchronize();
	std::chrono::high_resolution_clock::time_point t2 = std::chrono::high_resolution_clock::now();

	timePerBlock.increment(0, 5, (float) std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());

	// FB 3b
	hipDeviceSynchronize();
	t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION	

	// Resolution de l'OPF
	if (_nLine) {
		//std::cout << " Pn :" << std::endl;
		//Pn.display(true);
		//std::cout << " etaP :" << std::endl;
		//etaP.display(true);

		solveOPF();
		//std::cout << " Pso :" << std::endl;
		//Pso.display(true);
	}
	else {
		Pso = Pn;
		/*tempN1.set(&etaP);
		tempN1.divide(_rho1);
		Pso.add(&Pn);
		Pso.divide(2);
		Pso.subtract(&tempN1);*/
	}
	
	
	

#ifdef INSTRUMENTATION
	hipDeviceSynchronize();
	t2 = std::chrono::high_resolution_clock::now();
	timePerBlock.increment(0, 6, (float) std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());

	// FB 3c
	hipDeviceSynchronize();
	t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION

	// update Bp3
	updateEtaPBp3 << <_numBlocksN, _blockSize >> > (Bp3._matrixGPU, etaP._matrixGPU, nVoisin._matrixGPU, Pso._matrixGPU, Pn._matrixGPU, _rho1, _nAgent);
	updateLAMBDABt1GPU << <_numBlocksM, _blockSize >> > (Bt1._matrixGPU, LAMBDALin._matrixGPU, tradeLin._matrixGPU, _rhog, CoresLinTrans._matrixGPU, _nTrade);
	

#ifdef INSTRUMENTATION
	hipDeviceSynchronize();
	t2 = std::chrono::high_resolution_clock::now();
	timePerBlock.increment(0, 7, (float) std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
#endif // INSTRUMENTATION
}



float ADMMGPUConstCons3::updateResEndo(int iter)
{

	float resS = Tlocal.max2(&tradeLin);

	updateDiffGPU <<<_numBlocksM, _blockSize >> > (tempNN._matrixGPU, Tlocal._matrixGPU, CoresLinTrans._matrixGPU, _nAgent);
	float resR = tempNN.max2();

	float resXf = _ratioEps * Pso.max2(&Pn);
	resF.set(0, iter, resR);
	resF.set(1, iter, resS);
	resF.set(2, iter, resXf);
	return MYMAX(MYMAX(resXf, resS), resR);

}



