#include "hip/hip_runtime.h"
#include "../head/TestADMMGPUConst3.cuh"
#define NSTEPLOCAL 5
#define NMAXPEERPERTRHREAD 5

int testADMMGPUConst3()
{
	int n = 1;

	if (!testADMMGPUConst3Contruct1()) return n;
	n++;
	if (!testADMMGPUConst3Contruct2()) return n;
	n++;
	if (!testADMMGPUConst3Contruct3()) return n;
	n++;
	if (!testADMMGPUConst3LAMBDA()) return n;
	n++;
	if (!testADMMGPUConst3Bt1()) return n;
	n++;
	if (!testADMMGPUConst3TradeP()) return n;
	n++;
	if (!testADMMGPUConst3UpdateRes()) return n;
	n++;
	if (!testADMMGPUConst3CalcRes()) return n;
	n++;
	if (!testADMMGPUConst3Solve1()) return n;
	n++;
	if (!testADMMGPUConst3Solve2()) return n;
	n++;


	return 0;
}

void testADMMGPUConst3Time(int test) {

	switch (test)
	{
	case 0:
		testADMMGPUConst3TimeLAMBDA();
		break;
	case 1:
		testADMMGPUConst3TimeBt1();
		break;
	case 2:
		testADMMGPUConst3TimeTradeP();
		break;
	case 3:
		testADMMGPUConst3TimeUpdateRes();
		break;
	case 4:
		testADMMGPUConst3TimeCalcRes();
		break;
	default:
		std::cout << "No valid input " << std::endl;
		break;
	}
	
}

bool testADMMGPUConst3Contruct1()
{
	std::cout << "contructeur par defaut" << std::endl;
	ADMMGPUConst3 a;
	return true;
}

bool testADMMGPUConst3Contruct2()
{
	float rho = 2;

	std::cout << "contructeur avec parametres" << std::endl;
	ADMMGPUConst3 a(rho);
	return true;
}
bool testADMMGPUConst3Contruct3()
{
	float rho = 2;

	std::cout << "contructeur en deux temps" << std::endl;
	ADMMGPUConst3 a;
	a = ADMMGPUConst3(rho);
	return true;
}

bool testADMMGPUConst3Solve1()
{
	//solve(Simparam* result, Simparam sim, StudyCase cas);
	StudyCase cas;
	cas.Set2node();
	//cas.display();
	int nAgent = cas.getNagent();
	Simparam param(nAgent);
	param.setRho(1);
	Simparam res(param);

	ADMMGPUConst3 a;

	a.solve(&res, param, cas);
	res.display();
	MatrixCPU Trade(nAgent, nAgent);
	Trade.set(0, 1, -1);
	Trade.set(1, 0, 1);
	MatrixCPU Res(res.getRes());
	Res.display();

	MatrixCPU trade = res.getTrade();
	trade.display();
	return trade.isEqual(&Trade, 0.001);

}
bool testADMMGPUConst3Solve2()
{
	//solve(Simparam* result, Simparam sim, StudyCase cas);
	StudyCase cas;
	cas.Set29node();
	//cas.display();
	int nAgent = cas.getNagent();

	Simparam param(nAgent, cas.getNLine());
	float epsG = 0.00002f;
	float epsL = 0.000002f;
	param.setRho(10000);
	param.setEpsL(epsL);
	param.setEpsG(epsG);
	param.setStep(1, 1);
	Simparam res(param);
	ADMMGPUConst3 a;
	a.solve(&res, param, cas);
	res.display();


	float Pn[31] = { -1.008853555,-4.62966156,-2.927534103,-0.8979898691,-0.9462603927,-0.09805059433,-0.127968356,-4.168303013,-3.151874542,-2.261414766,-0.670329392,-3.399893284,-0.4841034412,-2.775528431,-3.008597374,-1.849177122,-0.5534118414,-2.362840891,-1.122991204,-0.1379692554,-2.332088947,4.406820297,5.406073093,3.676487684,3.929354668,4.570535183,2.529039145,3.478654861,2.755935192,3.768760443,4.393183708, };

	MatrixCPU P(31, 1);
	for (int i = 0; i < 31; i++) {
		P.set(i, 0, Pn[i]);
	}

	MatrixCPU P2 = res.getPn();
	P2.display();
	MatrixCPU temp(P2);
	temp.subtract(&P);
	temp.display();
	return P2.isEqual(&P, 0.01);

}

bool testADMMGPUConst3LAMBDA()
{
	int nAgent = 3; // 2 conso et un prod
	int ntrade = 4;
	int blockSize = 256;
	int numBlocks = ceil((nAgent + blockSize - 1) / blockSize);
	float value1 = 2;
	float value2 = -8;
	float value3 = 1.5;
	float value4 = 4;
	MatrixGPU Bt1(ntrade, 1, 0);
	MatrixGPU Bt11(ntrade, 1, -value4 - value1 / value3);
	MatrixGPU LAMBDALin(ntrade, 1, value1);
	MatrixGPU trade(ntrade, 1, value2);
	MatrixGPU CoresLinTrans(ntrade, 1);
	
	MatrixGPU LAMBDALin2(ntrade, 1, value1 + 0.5 * value3 * (value2 + value4));
	float rho = value3;


	CoresLinTrans.set(0, 0, 2);
	CoresLinTrans.set(1, 0, 3);
	CoresLinTrans.set(2, 0, 0);
	CoresLinTrans.set(3, 0, 1);

	trade.set(2, 0, value4);
	trade.set(3, 0, value4);

	
	trade.transferGPU();
	LAMBDALin.transferGPU();
	CoresLinTrans.transferGPU();
	
	updateLAMBDAGPU << <numBlocks, blockSize >> > (LAMBDALin._matrixGPU, trade._matrixGPU, rho, CoresLinTrans._matrixGPU, ntrade);

	
	LAMBDALin.transferCPU();

	return (LAMBDALin.isEqual(&LAMBDALin2));
}

bool testADMMGPUConst3Bt1()
{
	int nAgent = 3; // 2 conso et un prod
	int ntrade = 4;
	int blockSize = 256;
	int numBlocks = ceil((nAgent + blockSize - 1) / blockSize);
	float value1 = 2;
	float value2 = -8;
	float value3 = 1.5;
	float value4 = 4;
	MatrixGPU Bt1(ntrade, 1, 0);
	MatrixGPU Bt11(ntrade, 1, -value4 - value1 / value3);
	MatrixGPU trade(ntrade, 1, value2);
	MatrixGPU CoresLinTrans(ntrade, 1);

	MatrixGPU LAMBDALin(ntrade, 1, value1 + 0.5 * value3 * (value2 + value4));
	float rho = value3;


	CoresLinTrans.set(0, 0, 2);
	CoresLinTrans.set(1, 0, 3);
	CoresLinTrans.set(2, 0, 0);
	CoresLinTrans.set(3, 0, 1);

	trade.set(2, 0, value4);
	trade.set(3, 0, value4);
	Bt11.set(2, 0, -value2 - value1 / value3);
	Bt11.set(3, 0, -value2 - value1 / value3);



	Bt1.transferGPU();
	trade.transferGPU();
	LAMBDALin.transferGPU();

	CoresLinTrans.transferGPU();



	updateBt1GPU << <numBlocks, blockSize >> > (Bt1._matrixGPU, trade._matrixGPU, rho, LAMBDALin._matrixGPU, CoresLinTrans._matrixGPU, ntrade);

	Bt1.transferCPU();
	LAMBDALin.transferCPU();

	return Bt1.isEqual(&Bt11);
}

bool testADMMGPUConst3TradeP()
{
	int nAgent = 4;
	int ntrade = 8;
	int blockSize = 512;
	//int numBlocks = ceil((nAgent + blockSize - 1) / blockSize);
	float value1 = 2;
	float value2 = 3;
	float value3 = 1;
	float value4 = 1;
	float value5 = -1;
	float value6 = -30;
	float value7 = 10;
	float value8 = 5;
	float value9 = value8 - value3 + value4 - value5;
	MatrixGPU Bt1(ntrade, 1, value1);
	float at1 = value3;
	float at2 = value4;
	MatrixGPU Ct(ntrade, 1, value5);
	MatrixGPU Lb(ntrade, 1, value6);
	MatrixGPU Ub(ntrade, 1, value7);
	MatrixGPU Tlocal(ntrade, 1);
	
	MatrixGPU Tlocal_pre(ntrade, 1, value8);
	//Tlocal_pre.set(0,0,value8+1);
	MatrixGPU Tlocal2(Tlocal_pre);
	MatrixGPU CoresLinAgent(ntrade, 1);
	CoresLinAgent.set(1, 0, 0);
	CoresLinAgent.set(2, 0, 1);
	CoresLinAgent.set(3, 0, 1);
	CoresLinAgent.set(4, 0, 2);
	CoresLinAgent.set(5, 0, 2);
	CoresLinAgent.set(6, 0, 3);
	CoresLinAgent.set(7, 0, 3);
	MatrixGPU Bp1(nAgent, 1);
	MatrixGPU Bp11(nAgent, 1);
	MatrixGPU Ap1(nAgent, 1, value2);
	MatrixGPU Ap2(nAgent, 1, value3);
	MatrixGPU Ap12(nAgent, 1, value2 + value3);
	MatrixGPU Cp(nAgent, 1, value5);
	MatrixGPU PLb(nAgent, 1, value6/2);
	MatrixGPU PUb(nAgent, 1, value7/2);

	MatrixGPU nVoisin(nAgent, 1, 2);
	MatrixGPU Tmoy(nAgent, 1, value3);
	MatrixGPU Tmoy2(Tmoy);
	MatrixGPU MU(nAgent, 1, value5);
	MatrixGPU MU2(MU);
	MatrixGPU P(nAgent, 1, value4);
	MatrixGPU P2(P);

	MatrixGPU CoresAgentLin(nAgent + 1, 1);
	CoresAgentLin.set(1, 0, 2);
	CoresAgentLin.set(2, 0, 4);
	CoresAgentLin.set(3, 0, 6);
	CoresAgentLin.set(4, 0, 8);

	for (int iter = 0; iter < NSTEPLOCAL; iter++) {
		for (int i = 0; i < nAgent; i += 1) // 1 bloc = 1 agent
		{
			float s = 0.0;
			for (int j = CoresAgentLin.get(i, 0); j < CoresAgentLin.get(i + 1, 0); j++) // on parcourt les trades de l'agent i
			{
				float m = Tlocal2.get(j, 0) - Tmoy2.get(i, 0) + P2.get(i, 0) - MU2.get(i, 0);
				float r = (Bt1.get(j, 0) * at1 + m * at2 - Ct.get(j, 0)) / (at1 + at2);
				float ub = Ub.get(j, 0);
				float lb = Lb.get(j, 0);
				float t = (ub - r) * (r > ub) + (lb - r) * (r < lb) + r;
				Tlocal2.set(j, 0, t);
				s += t;
			}
			float r = s / nVoisin.get(i, 0);
			Tmoy2.set(i, 0, r);
			Bp11.set(i, 0, r + MU2.get(i, 0));
			float p = (Ap1.get(i, 0) * Bp11.get(i, 0) - Cp.get(i, 0)) / (Ap12.get(i, 0));
			float ub = PUb.get(i, 0);
			float lb = PLb.get(i, 0);
			p = (ub - p) * (p > ub) + (lb - p) * (p < lb) + p;
			P2.set(i, 0, p);
			MU2.set(i, 0, MU2.get(i, 0) + r - P2.get(i, 0));
		}
	}
	


	Bt1.transferGPU();
	Tlocal.transferGPU();
	Ct.transferGPU();
	Lb.transferGPU();
	Ub.transferGPU();
	Tlocal_pre.transferGPU();
	Tmoy.transferGPU();
	P.transferGPU();
	MU.transferGPU();
	CoresLinAgent.transferGPU();
	Ap1.transferGPU();
	Ap2.transferGPU();
	Ap12.transferGPU();
	Bp1.transferGPU();
	Cp.transferGPU();
	PLb.transferGPU();
	PUb.transferGPU();
	nVoisin.transferGPU();
	CoresAgentLin.transferGPU();

	std::cout << " fin transfert " << std::endl;


	updateTradePGPUShared<256> <<<nAgent, blockSize >> > (Tlocal._matrixGPU, Tlocal_pre._matrixGPU, Tmoy._matrixGPU, P._matrixGPU, MU._matrixGPU, nVoisin._matrixGPU, at1, at2, Bt1._matrixGPU, Ct._matrixGPU,
		Lb._matrixGPU, Ub._matrixGPU, Ap1._matrixGPU, Ap12._matrixGPU, Cp._matrixGPU, PLb._matrixGPU, PUb._matrixGPU, CoresAgentLin._matrixGPU);

	Tlocal.transferCPU();
	P.transferCPU();
	MU.transferCPU();
	Tmoy.transferCPU();
	
	Tlocal2.display();
	Tlocal.display();
	Tmoy2.display();
	Tmoy.display();
	P2.display();
	P.display();
	MU2.display();
	MU.display();

	Bp11.display();


	return ((Tlocal.isEqual(&Tlocal2)) && (P.isEqual(&P2)) && (MU.isEqual(&MU2)) && (Tmoy.isEqual(&Tmoy2)));
}

bool testADMMGPUConst3UpdateRes()
{	/*float ADMMGPU5::updateRes(MatrixCPU* res, MatrixGPU* Tlocal, MatrixGPU* trade, int iter, MatrixGPU* CoresLinAgent, MatrixGPU* CoresLinVoisin, MatrixGPU* CoresMatLin, MatrixGPU* tempNN)
{
	tempNN->subtract(Tlocal, trade);
	//hipDeviceSynchronize();
	float resS = tempNN->distance2();

	updateDiffGPU <<<_numBlocks, _blockSize >>> (tempNN->_matrixGPU, Tlocal->_matrixGPU, CoresLinTrans->_matrixGPU, _N);
	//hipDeviceSynchronize();
	float resR = tempNN->distance2();

	res->set(0, iter, resR);
	res->set(1, iter, resS);


	return resR * (resR > resS) + resS * (resR <= resS);*/
	int nAgent = 3;
	int ntrade = 4;
	int blockSize = 15;
	int numBlocks = ceil((ntrade + blockSize - 1) / blockSize);
	std::cout << "blockSize " << blockSize << " numBlocks " << numBlocks << std::endl;
	float value1 = 4;
	float value2 = 2.5;
	float value3 = -2;
	float value4 = value3 + value2 - value1;
	MatrixCPU res(2, 1);
	MatrixCPU res2(2, 1);
	MatrixGPU Tlocal(ntrade, 1, value1);
	MatrixGPU Tlocal_pre(ntrade, 1, value2);

	MatrixGPU CoresLinTrans(ntrade, 1);

	MatrixGPU tempN(numBlocks, 1, 0, 1);
	MatrixGPU tempN2(numBlocks, 1, 0, 1);
	
	

	CoresLinTrans.set(0, 0, 2);
	CoresLinTrans.set(1, 0, 3);
	CoresLinTrans.set(2, 0, 0);
	CoresLinTrans.set(3, 0, 1);

	Tlocal.set(2, 0, value3);
	Tlocal.set(3, 0, value3);
	Tlocal_pre.set(2, 0, value4);
	Tlocal_pre.set(3, 0, value4);



	res2.set(0, 0, sqrtf((value1 + value3) * (value1 + value3) ));
	res2.set(1, 0, sqrtf((value1 - value2) * (value1 - value2) ));
	int iter = 0;

	Tlocal.transferGPU();
	Tlocal_pre.transferGPU();
	CoresLinTrans.transferGPU();
	
	float resS = Tlocal.max2(&Tlocal_pre);
	updateDiffGPU << <numBlocks, blockSize >> > (tempN._matrixGPU, Tlocal._matrixGPU, CoresLinTrans._matrixGPU, ntrade);
	float resR = tempN.max2();
	
	res.set(0, 0, resR);
	res.set(1, 0, resS);

	return res2.isEqual(&res);
}
bool testADMMGPUConst3CalcRes() {
	/*
	* float ADMMGPU5::calcRes( MatrixGPU* Tlocal, MatrixGPU* P, MatrixGPU* tempN1, MatrixGPU* tempNN)
{
	 tempNN->subtract(Tlocal, &Tlocal_pre);
	 tempN1->subtract(&Tmoy, P);

	 float d1 = tempN1->max2();
	 float d2 = tempNN->max2();


	 return d1* (d1 > d2) + d2 * (d2 >= d1);
}*/

	int nAgent = 3;
	int ntrade = 4;
	int blockSize = 256;
	int numBlocks = ceil((ntrade + blockSize - 1) / blockSize);
	float value1 = 5;
	float value2 = 2;
	float value3 = -3;
	float value4 = -1;
	MatrixGPU Tlocal(ntrade, 1, value1);
	MatrixGPU Tlocal_pre(ntrade, 1, value2);
	MatrixGPU Tmoy(nAgent, 1, value3);
	MatrixGPU P(nAgent, 1, value4);


	Tlocal.transferGPU();
	Tlocal_pre.transferGPU();
	Tmoy.transferGPU();
	P.transferGPU();
	

	float d11 = Tlocal.max2(&Tlocal_pre);
	float d22 = P.max2(&Tmoy);
	float d = d11 * (d11 > d22) + d22 * (d22 >= d11);



	float d1 = fabs(value1 - value2);
	float d2 = fabs(value3 - value4);
	float df = d1 * (d1 > d2) + d2 * (d2 >= d1);



	return (df == d);
}


void testADMMGPUConst3TimeLAMBDA()
{
	std::string fileName = "TempsLAMBDA.csv";
	//hipEvent_t start, stop;
	//float elapsedTime;
	std::chrono::high_resolution_clock::time_point a;
	std::chrono::high_resolution_clock::time_point b;
	unsigned int time;
	std::ios_base::openmode mode = std::fstream::in | std::fstream::out | std::fstream::app;
	const int nNAgent = 7;
	const int nSimu = 100;
	const int nRepet = 10;
	int nAgent[nNAgent] = { 10, 100, 500, 1000, 5000, 10000, 40000 }; // autant conso que de prod, la derni�re veleur ne "marche" pas (trop rapide)
	int ntrade[nNAgent];
	int blockSize = 256;
	float values1[nSimu];
	float values2[nSimu];
	float rhos[nSimu];
	MatrixCPU temps(nNAgent, nSimu, 0);
	
	for (int j = 0; j < nSimu;j++) {
		values1[j] = (float) (rand()) / rand();
		values2[j] = (float)(rand()) / rand();
		rhos[j] = (float)(rand()%100) / rand();
	}

	for (int i = 0; i < nNAgent; i++) {
		
		ntrade[i] = nAgent[i] * nAgent[i] / 2;
		std::cout << "iteration " << i << " nAgent " << nAgent[i] << " ntrade " << ntrade[i] << std::endl;
		int numBlocks = ceil((ntrade[i] + blockSize - 1) / blockSize);
		
		MatrixGPU CoresLinAgent(ntrade[i], 1);
		MatrixGPU CoresLinVoisin(ntrade[i], 1);
		MatrixGPU CoresMatLin(nAgent[i], nAgent[i], -1);
		MatrixGPU CoresLinTrans(ntrade[i], 1);
		

		int indice = 0;
		int idVoisin = 0;

		for (int idAgent = 0; idAgent < nAgent[i]; idAgent++) {
			int Nvoisinmax = nAgent[i] / 2;
			if (idAgent < Nvoisinmax) {
				idVoisin = 0;
			}
			else {
				idVoisin = nAgent[i] / 2;
			}
			for (int voisin = idVoisin; voisin < Nvoisinmax; voisin++) {
				CoresLinAgent.set(indice, 0, idAgent);
				CoresLinVoisin.set(indice, 0, voisin);
				CoresMatLin.set(idAgent, voisin, indice);
				indice = indice + 1;
			}
		}
		for (int lin = 0;lin < ntrade[i];lin++) {
			int i = CoresLinAgent.get(lin, 0);
			int j = CoresLinVoisin.get(lin, 0);
			int k = CoresMatLin.get(j, i);
			CoresLinTrans.set(lin, 0, k);
		}
		CoresLinAgent.transferGPU();
		CoresLinVoisin.transferGPU();
		CoresMatLin.transferGPU();
		CoresLinTrans.transferGPU();
		clock_t t = clock();
		
		for (int simu = 0; simu < nSimu; simu++) {
			MatrixGPU LAMBDALin(ntrade[i], 1, values1[simu], 1);
			MatrixGPU trade(ntrade[i], 1, values2[simu], 1);
			float rho = rhos[simu];
			/*hipEventCreate(&start);
			hipEventRecord(start, 0);*/
			hipDeviceSynchronize();
			time = 0;
			for (int repet = 0; repet < nRepet; repet++) {
				MatrixGPU LAMBDALinCopy(LAMBDALin);
				MatrixGPU tradeCopy(trade);
				MatrixGPU CoresLinTransCopy(CoresLinTrans);
				hipDeviceSynchronize();
				a = std::chrono::high_resolution_clock::now();
				updateLAMBDAGPU <<<numBlocks, blockSize >>> (LAMBDALinCopy._matrixGPU, tradeCopy._matrixGPU, rho, CoresLinTransCopy._matrixGPU, ntrade[i]);
				hipDeviceSynchronize();
				b = std::chrono::high_resolution_clock::now();
				time += std::chrono::duration_cast<std::chrono::nanoseconds>(b - a).count();
			}
			temps.set(i, simu, (float) time/nRepet);
		}
	}
	temps.saveCSV(fileName, mode);
}

void testADMMGPUConst3TimeBt1()
{
	std::string fileName = "TempsBt.csv";
	//hipEvent_t start, stop;
	//float elapsedTime;
	std::chrono::high_resolution_clock::time_point a;
	std::chrono::high_resolution_clock::time_point b;
	unsigned int time;
	std::ios_base::openmode mode = std::fstream::in | std::fstream::out | std::fstream::app;
	const int nNAgent = 7;
	const int nSimu = 100;
	const int nRepet = 10;
	int nAgent[nNAgent] = { 10, 100, 500, 1000, 5000, 10000, 40000 }; // autant conso que de prod, la derni�re veleur ne "marche" pas (trop rapide)
	int ntrade[nNAgent];
	int blockSize = 256;
	float values1[nSimu];
	float values2[nSimu];
	float rhos[nSimu];
	MatrixCPU temps(nNAgent, nSimu, 0);

	for (int j = 0; j < nSimu;j++) {
		values1[j] = (float)(rand()) / rand();
		values2[j] = (float)(rand()) / rand();
		rhos[j] = (float)(rand() % 100) / rand();
	}

	for (int i = 0; i < nNAgent; i++) {

		ntrade[i] = nAgent[i] * nAgent[i] / 2;
		std::cout << "iteration " << i << " nAgent " << nAgent[i] << " ntrade " << ntrade[i] << std::endl;
		int numBlocks = ceil((ntrade[i] + blockSize - 1) / blockSize);

		MatrixGPU Bt1(ntrade[i], 1, 0, 1);

		MatrixGPU CoresLinAgent(ntrade[i], 1);
		MatrixGPU CoresLinVoisin(ntrade[i], 1);
		MatrixGPU CoresMatLin(nAgent[i], nAgent[i], -1);
		MatrixGPU CoresLinTrans(ntrade[i], 1);


		int indice = 0;
		int idVoisin = 0;

		for (int idAgent = 0; idAgent < nAgent[i]; idAgent++) {
			int Nvoisinmax = nAgent[i] / 2;
			if (idAgent < Nvoisinmax) {
				idVoisin = 0;
			}
			else {
				idVoisin = nAgent[i] / 2;
			}
			for (int voisin = idVoisin; voisin < Nvoisinmax; voisin++) {
				CoresLinAgent.set(indice, 0, idAgent);
				CoresLinVoisin.set(indice, 0, voisin);
				CoresMatLin.set(idAgent, voisin, indice);
				indice = indice + 1;
			}
		}
		for (int lin = 0;lin < ntrade[i];lin++) {
			int i = CoresLinAgent.get(lin, 0);
			int j = CoresLinVoisin.get(lin, 0);
			int k = CoresMatLin.get(j, i);
			CoresLinTrans.set(lin, 0, k);
		}
		CoresLinAgent.transferGPU();
		CoresLinVoisin.transferGPU();
		CoresMatLin.transferGPU();
		CoresLinTrans.transferGPU();
		clock_t t = clock();

		for (int simu = 0; simu < nSimu; simu++) {
			MatrixGPU LAMBDALin(ntrade[i], 1, values1[simu], 1);
			MatrixGPU trade(ntrade[i], 1, values2[simu], 1);
			float rho = rhos[simu];
			/*hipEventCreate(&start);
			hipEventRecord(start, 0);*/
			hipDeviceSynchronize();
			time = 0;
			for (int repet = 0; repet < nRepet; repet++) {
				MatrixGPU LAMBDALinCopy(LAMBDALin);
				MatrixGPU tradeCopy(trade);
				MatrixGPU CoresLinTransCopy(CoresLinTrans);
				MatrixGPU Bt1Copy(Bt1);
				hipDeviceSynchronize();
				a = std::chrono::high_resolution_clock::now();
				updateBt1GPU << <numBlocks, blockSize >> > (Bt1Copy._matrixGPU, tradeCopy._matrixGPU, rho, LAMBDALinCopy._matrixGPU, CoresLinTransCopy._matrixGPU, ntrade[i]);
				hipDeviceSynchronize();
				b = std::chrono::high_resolution_clock::now();
				time += std::chrono::duration_cast<std::chrono::nanoseconds>(b - a).count();
			}
			temps.set(i, simu, (float)time / nRepet);
		}
	}
	temps.saveCSV(fileName, mode);
}


void testADMMGPUConst3TimeTradeP() {
	std::string fileName = "TempsTradeP10_F5.csv";
	std::chrono::high_resolution_clock::time_point a;
	std::chrono::high_resolution_clock::time_point b;
	unsigned int time;
	std::ios_base::openmode mode = std::fstream::in | std::fstream::out | std::fstream::app;
	const int nNAgent = 6;
	const int nSimu = 100;
	const int nRepet = 10;
	const int nVar = 14;
	int nAgent[nNAgent] = { 10, 100, 500, 1000, 5000, 10000 }; // autant conso que de prod, 
	int ntrade[nNAgent];
	int blockSize = 256;
	float values[nVar][nSimu];
	float rhos[nSimu];
	MatrixCPU temps(nSimu, nNAgent, 0);
	MatrixCPU nAgentMat(1, nNAgent, 0);
	MatrixCPU nTradeMat(1, nNAgent, 0);
	MatrixCPU nPro(1, nNAgent, 0);

	for (int j = 0; j < nSimu; j++) {
		for (int var = 0; var < nVar; var++) {
			values[var][j] = (float)(rand()) / rand();
		}
		rhos[j] = (float)(rand() % 100) / rand();
	}

	for (int i = 0; i < nNAgent; i++) {

		ntrade[i] = nAgent[i] * nAgent[i] / 2;
		std::cout << "iteration " << i << " nAgent " << nAgent[i] << " ntrade " << ntrade[i] << std::endl;
		int numBlocks = nAgent[i];
		nAgentMat.set(0, i, nAgent[i]);
		nTradeMat.set(0, i, ntrade[i]);
		nPro.set(0, i, numBlocks * blockSize);
		MatrixGPU Tlocal(ntrade[i], 1, 0, 1);
		MatrixGPU CoresLinVoisin(ntrade[i], 1);
		MatrixGPU CoresAgentLin(nAgent[i] + 1, 1);

		int indice = 0;
		int idVoisin = 0;

		for (int idAgent = 0; idAgent < nAgent[i]; idAgent++) {
			int Nvoisinmax = nAgent[i] / 2;
			if (idAgent < Nvoisinmax) {
				idVoisin = 0;
			}
			else {
				idVoisin = nAgent[i] / 2;
			}
			for (int voisin = idVoisin; voisin < Nvoisinmax; voisin++) {
				CoresLinVoisin.set(indice, 0, voisin);

				indice = indice + 1;
			}
			CoresAgentLin.set(idAgent + 1, 0, indice);


		}
		CoresLinVoisin.transferGPU();
		CoresAgentLin.transferGPU();
		clock_t t = clock();

		for (int simu = 0; simu < nSimu; simu++) {

			MatrixGPU Tlocal_pre(ntrade[i], 1, values[0][simu], 1);
			MatrixGPU Bt1(ntrade[i], 1, values[1][simu], 1);
			MatrixGPU Ct(ntrade[i], 1, values[2][simu], 1);
			MatrixGPU Lb(ntrade[i], 1, values[3][simu], 1);
			MatrixGPU Ub(ntrade[i], 1, values[4][simu], 1);

			MatrixGPU Tmoy(nAgent[i], 1, values[5][simu], 1);
			MatrixGPU P(nAgent[i], 1, values[6][simu], 1);
			MatrixGPU MU(nAgent[i], 1, values[7][simu], 1);
			MatrixGPU nVoisin(nAgent[i], 1, nAgent[i] / 2, 1);
			MatrixGPU Ap1(nAgent[i], 1, values[9][simu], 1);
			MatrixGPU Ap12(nAgent[i], 1, values[10][simu], 1);
			MatrixGPU Cp(nAgent[i], 1, values[11][simu], 1);
			MatrixGPU PLb(nAgent[i], 1, values[12][simu], 1);
			MatrixGPU PUb(nAgent[i], 1, values[13][simu], 1);

			float rho = rhos[simu];
			float at1 = 2 * rho;
			float at2 = 3 * rho;


			time = 0;
			for (int repet = 0; repet < nRepet; repet++) {
				int _n = nAgent[i];
				MatrixGPU Tlocal_preCopy(Tlocal_pre);
				MatrixGPU Bt1Copy(Bt1);
				MatrixGPU CtCopy(Ct);
				MatrixGPU LbCopy(Lb);
				MatrixGPU UbCopy(Ub);

				MatrixGPU TmoyCopy(Tmoy);
				MatrixGPU PCopy(P);
				MatrixGPU MUCopy(MU);
				MatrixGPU nVoisinCopy(nVoisin);
				MatrixGPU Ap1Copy(Ap1);
				MatrixGPU Ap12Copy(Ap12);
				MatrixGPU CpCopy(Cp);
				MatrixGPU PLbCopy(PLb);
				MatrixGPU PUbCopy(PUb);

				MatrixGPU TlocalCopy(Tlocal);
				MatrixGPU CoresLinVoisinCopy(CoresLinVoisin);
				MatrixGPU CoresAgentLinCopy(CoresAgentLin);

				hipDeviceSynchronize();
				a = std::chrono::high_resolution_clock::now();
				updateTradePGPUShared<256> << <_n, blockSize >> > (TlocalCopy._matrixGPU, Tlocal_preCopy._matrixGPU, TmoyCopy._matrixGPU, PCopy._matrixGPU,
					MUCopy._matrixGPU, nVoisinCopy._matrixGPU, at1, at2, Bt1Copy._matrixGPU, CtCopy._matrixGPU, LbCopy._matrixGPU, UbCopy._matrixGPU,
					Ap1Copy._matrixGPU, Ap12Copy._matrixGPU, CpCopy._matrixGPU, PLbCopy._matrixGPU, PUbCopy._matrixGPU, CoresAgentLinCopy._matrixGPU);
				hipDeviceSynchronize();
				b = std::chrono::high_resolution_clock::now();
				time += std::chrono::duration_cast<std::chrono::nanoseconds>(b - a).count();
			}
			temps.set(simu, i, (float)time / nRepet);
		}
	}

	nAgentMat.saveCSV(fileName, mode);
	nTradeMat.saveCSV(fileName, mode);
	nPro.saveCSV(fileName, mode);
	temps.saveCSV(fileName, mode);
}



void testADMMGPUConst3TimeUpdateRes() {
	std::string fileName = "TempsResG.csv";
	
	std::chrono::high_resolution_clock::time_point a;
	std::chrono::high_resolution_clock::time_point b;
	unsigned int time;
	std::ios_base::openmode mode = std::fstream::in | std::fstream::out | std::fstream::app;
	const int nNAgent = 7;
	const int nSimu = 100;
	const int nRepet = 10;
	int nAgent[nNAgent] = { 10, 100, 500, 1000, 5000, 10000, 40000 }; // autant conso que de prod, la derni�re veleur ne "marche" pas (trop rapide)
	int ntrade[nNAgent];
	int blockSize = 256;
	float values1[nSimu];
	float values2[nSimu];
	
	MatrixCPU temps(nNAgent, nSimu, 0);
	MatrixCPU res(2, 1, 0);

	for (int j = 0; j < nSimu;j++) {
		values1[j] = (float)(rand()) / rand();
		values2[j] = (float)(rand()) / rand();
	}

	for (int i = 0; i < nNAgent; i++) {

		ntrade[i] = nAgent[i] * nAgent[i] / 2;
		std::cout << "iteration " << i << " nAgent " << nAgent[i] << " ntrade " << ntrade[i] << std::endl;
		int numBlocks = ceil((ntrade[i] + blockSize - 1) / blockSize);

		MatrixGPU tempN(blockSize, 1, 0, 1);
		MatrixGPU tempN2(blockSize, 1, 0, 1);

		MatrixGPU CoresLinAgent(ntrade[i], 1);
		MatrixGPU CoresLinVoisin(ntrade[i], 1);
		MatrixGPU CoresMatLin(nAgent[i], nAgent[i], -1);
		MatrixGPU CoresLinTrans(ntrade[i], 1);


		int indice = 0;
		int idVoisin = 0;

		for (int idAgent = 0; idAgent < nAgent[i]; idAgent++) {
			int Nvoisinmax = nAgent[i] / 2;
			if (idAgent < Nvoisinmax) {
				idVoisin = 0;
			}
			else {
				idVoisin = nAgent[i] / 2;
			}
			for (int voisin = idVoisin; voisin < Nvoisinmax; voisin++) {
				CoresLinAgent.set(indice, 0, idAgent);
				CoresLinVoisin.set(indice, 0, voisin);
				CoresMatLin.set(idAgent, voisin, indice);
				indice = indice + 1;
			}
		}
		for (int lin = 0;lin < ntrade[i];lin++) {
			int i = CoresLinAgent.get(lin, 0);
			int j = CoresLinVoisin.get(lin, 0);
			int k = CoresMatLin.get(j, i);
			CoresLinTrans.set(lin, 0, k);
		}
		CoresLinAgent.transferGPU();
		CoresLinVoisin.transferGPU();
		CoresMatLin.transferGPU();
		CoresLinTrans.transferGPU();
		clock_t t = clock();

		for (int simu = 0; simu < nSimu; simu++) {
			MatrixGPU Tlocal_pre(ntrade[i], 1, values1[simu], 1);
			MatrixGPU Tlocal(ntrade[i], 1, values2[simu], 1);
			
			hipDeviceSynchronize();
			time = 0;
			for (int repet = 0; repet < nRepet; repet++) {
				MatrixGPU Tlocal_preCopy(Tlocal_pre);
				MatrixGPU TlocalCopy(Tlocal);
				MatrixGPU CoresLinTransCopy(CoresLinTrans);
				MatrixGPU tempNCopy(tempN);
				MatrixGPU tempN2Copy(tempN2);
				hipDeviceSynchronize();
				a = std::chrono::high_resolution_clock::now();
				float resS = TlocalCopy.max2(&Tlocal_preCopy);
				updateDiffGPU << <numBlocks, blockSize >> > (tempNCopy._matrixGPU, TlocalCopy._matrixGPU, CoresLinTransCopy._matrixGPU, ntrade[i]);
				float resR = tempNCopy.max2();
				hipDeviceSynchronize();
				b = std::chrono::high_resolution_clock::now();
				time += std::chrono::duration_cast<std::chrono::nanoseconds>(b - a).count();
			}
			temps.set(i, simu, (float)time / nRepet);
		}
	}
	temps.saveCSV(fileName, mode);

}

void testADMMGPUConst3TimeCalcRes() {
	std::string fileName = "TempsResL.csv";

	std::chrono::high_resolution_clock::time_point a;
	std::chrono::high_resolution_clock::time_point b;
	unsigned int time;
	std::ios_base::openmode mode = std::fstream::in | std::fstream::out | std::fstream::app;
	const int nNAgent = 7;
	const int nSimu = 100;
	const int nRepet = 10;
	int nAgent[nNAgent] = { 10, 100, 500, 1000, 5000, 10000, 40000 }; // autant conso que de prod, la derni�re veleur ne "marche" pas (trop rapide)
	int ntrade[nNAgent];
	int blockSize = 256;
	float values1[nSimu];
	float values2[nSimu];

	MatrixCPU temps(nNAgent, nSimu, 0);
	
	for (int j = 0; j < nSimu;j++) {
		values1[j] = (float)(rand()) / rand();
		values2[j] = (float)(rand()) / rand();
	}

	for (int i = 0; i < nNAgent; i++) {

		ntrade[i] = nAgent[i] * nAgent[i] / 2;
		std::cout << "iteration " << i << " nAgent " << nAgent[i] << " ntrade " << ntrade[i] << std::endl;
		int numBlocks = ceil((ntrade[i] + blockSize - 1) / blockSize);

		MatrixGPU tempN(blockSize, 1, 0, 1);
		MatrixGPU tempN2(blockSize, 1, 0, 1);

		clock_t t = clock();

		for (int simu = 0; simu < nSimu; simu++) {
			MatrixGPU Tlocal_pre(ntrade[i], 1, values1[simu], 1);
			MatrixGPU Tlocal(ntrade[i], 1, values2[simu], 1);
			MatrixGPU P(nAgent[i], 1, values1[simu], 1);
			MatrixGPU Tmoy(nAgent[i], 1, values2[simu], 1);

			hipDeviceSynchronize();
			time = 0;
			for (int repet = 0; repet < nRepet; repet++) {
				MatrixGPU Tlocal_preCopy(Tlocal_pre);
				MatrixGPU TlocalCopy(Tlocal);
				MatrixGPU PCopy(P);
				MatrixGPU TmoyCopy(Tmoy);
				MatrixGPU tempNCopy(tempN);
				MatrixGPU tempN2Copy(tempN2);
				hipDeviceSynchronize();
				a = std::chrono::high_resolution_clock::now();
				
				float d11 = Tlocal.max2(&Tlocal_pre);
				float d22 = P.max2(&Tmoy);
			
				hipDeviceSynchronize();
				b = std::chrono::high_resolution_clock::now();
				time += std::chrono::duration_cast<std::chrono::nanoseconds>(b - a).count();
			}


			/*hipEventCreate(&stop);
			hipEventRecord(stop, 0);
			hipEventSynchronize(stop);
			hipEventElapsedTime(&elapsedTime, start, stop);*/
			temps.set(i, simu, (float)time / nRepet);
		}
	}
	temps.saveCSV(fileName, mode);

}



