#include "hip/hip_runtime.h"
﻿#include "../head/GPUPFdistPQ.cuh"


GPUPFdistPQ::GPUPFdistPQ() {}
GPUPFdistPQ::~GPUPFdistPQ() {}

void GPUPFdistPQ::init(const StudyCase& cas, MatrixGPU* PQ)
{
#ifdef INSTRUMENTATION
    t1 = std::chrono::high_resolution_clock::now();
    timePerBlock = MatrixCPU(1, 9); // Fb0 : init, Fb1ab : Flu, Fb2abc: Tension , FB3 : puissance, Fb4 erreur, Fb0 mise à jour

    occurencePerBlock = MatrixCPU(1, 9);; //nb de fois utilisé pendant la simu

#endif // INSTRUMENTATION
    

    //std::cout << "init PF GPUPFdistPQ" <<std::endl;
    //PQ->display(true);
    Nagent = cas.getNagent();
    Nbus = cas.getNBus();
    B2 = 2 * Nbus;
    N2 = 2 * Nagent;
   
    Nline = cas.getNLine(true); // ne doit pas être réduit ici !!!
    BL2 = Nbus + 2 * Nline;
    Nconstraint = B2 + Nline;
    //std::cout << Nline << " " << Nbus << std::endl;
    iterM = 30;
    iter = 0;
    epsPF = 0.00005;
    status = 0;
    numBlock = ceil((Nbus + _blockSize - 1) / _blockSize);
    //std::cout << numBlock << " " << _blockSize << std::endl;
    if (Nbus > 1024) {
        throw std::invalid_argument("too much bus, must change the computation of the voltage and S");
    }
    //CHECK_LAST_CUDA_ERROR();
    V0 = cas.getV0();
    theta0 = cas.gettheta0();

    //std::cout << "V0 :" << V0 << " theta0 " << theta0 << std::endl;
    v0 = V0 * cos(theta0);
    w0 = V0 * sin(theta0);
    _name = "Power summation method GPU"; // meilleure covergence quand c'est beaucoup chargé

    W0 = MatrixGPU(B2, 1, 0, 1);
    CoresAgentBus = MatrixGPU(cas.getCoresAgentBusLin(), 1);
    CoresAgentBusBegin = MatrixGPU(cas.getCoresAgentBusLinBegin(), 1);
    NagentByBus = MatrixGPU(cas.getNagentByBus(), 1);

    removeLossAgent << <1, 1 >> > (NagentByBus._matrixGPU, CoresAgentBusBegin._matrixGPU);

    //CHECK_LAST_CUDA_ERROR();

    calculW0Bis(PQ);
    //CHECK_LAST_CUDA_ERROR();
    /*std::cout << " W0 : " << std::endl;
    W0.display(true);*/
    Y = MatrixGPU(2 * Nbus + Nline, 1, 0, 1);
    W = MatrixGPU(B2, 1, 0, 1);
   
    W.preallocateReduction(); // calcul des pertes
   
    dW = MatrixGPU(B2, 1, 0, 1);
    dW.preallocateReduction();
   
    _Pintermediate = MatrixGPU(BL2, 1, 0, 1);
    
    _Qintermediate = MatrixGPU(BL2, 1, 0, 1);
    E = MatrixGPU(B2, 1, 0, 1);
   
    VoltageRealIm = MatrixGPU(B2, 1, 0, 1);
    
    VoltageRealImPre = MatrixGPU(B2, 1, 0, 1);
    
    VoltageRealImPre.preallocateReduction();
    

    initE << <numBlock, _blockSize >> > (E._matrixGPU, theta0, V0, Nbus);
    initECar << <numBlock, _blockSize >> > (VoltageRealIm._matrixGPU, v0, w0, Nbus);
    
    //VoltageRealIm.display(true);

    // W0[2 * N] : puissance active et réactive au noeud (I*[P Q])
    // W[2 * N] : puissance obtenue par calcul à partir de E
    // dW[2 * N] : derive de puissance
    // E[2 * N] : angle puis tension [O et 1] pour l'init ?
    // dE[2 * N] : derive de angle puis tension
   
  
    CoresLineBus = MatrixGPU(cas.getCoresLineBus(true));
    _CoresVoiLin = MatrixGPU(cas.getCoresVoiLin(), 1);
    _CoresBusLin = MatrixGPU(cas.getCoresBusLin(), 1);
    _nLines = MatrixGPU(cas.getNLines(), 1);
    
    //CHECK_LAST_CUDA_ERROR();


    //Bgrid = cas.getLineSuceptance();
    //Ggrid = cas.getLineReactance();
    _Blin = MatrixGPU(cas.getBlin(), 1);
    _Glin = MatrixGPU(cas.getGlin(), 1);
    _Blin2 = MatrixGPU(cas.getBlin2(), 1);
    _Glin2 = MatrixGPU(cas.getGlin2(), 1);


    // specificite algo
    // CoresLineBus.display();
    //std::cout << Nbus << " " << Nline << std::endl;
    ZsRe = MatrixGPU(cas.getZsRe(), 1);
    ZsIm = MatrixGPU(cas.getZsImag(), 1);
    Yd = MatrixGPU(cas.getYd(), 1);
    chekcase();

    F = MatrixGPU(Nbus, 1, -1); // F_i = bus antécédent de i
    nChild = MatrixGPU(Nbus, 1, 0);
    CoresLineBusGPU = MatrixGPU(2, Nline);
    F.set(1, 0, 0);
    if (Nbus != (Nline + 1)) {
        std::cout << "Warning this is not a distribution network, F not set" << std::endl;
    
    }
    else {
        for (int lold = 0; lold < Nline; lold++) {
            int busTo = CoresLineBus.get(lold, 1);
            int busFrom = CoresLineBus.get(lold, 0);
            F.set(busTo, 0, busFrom);
            nChild.set(busFrom, 0, nChild.get(busFrom, 0) + 1);
            CoresLineBusGPU.set(0, lold, busFrom);
            CoresLineBusGPU.set(1, lold, busTo);
        }
    } 
    LastBus = cas.getLastBus(); 
    //CoresLineBus.display();
    //std::cout << "LastBus " << LastBus << std::endl;
    //CHECK_LAST_CUDA_ERROR();
    int debutChild = 0;
    MatrixCPU nChildTemp(Nbus, 1, 0);
    _indiceChildBegin = MatrixGPU(Nline, 1);
    Childs = MatrixGPU(Nbus, 1);
    for (int i = 0; i < Nbus; i++) {
        if (i > 0) {
            _indiceChildBegin.set(i - 1, 0, debutChild);

            int Ai = F.get(i, 0);
            Childs.set(_indiceChildBegin.get(Ai, 0) + nChildTemp.get(Ai, 0), 0, i);
            nChildTemp.increment(Ai, 0, 1);
            debutChild += nChild.get(i - 1, 0);
        }

    }
  
    //CoresLineBus.transferGPU();
    CoresLineBusGPU.transferGPU();
    F.transferGPU();
    nChild.transferGPU();
    Childs.transferGPU();
    _indiceChildBegin.transferGPU();
    
   
    //CHECK_LAST_CUDA_ERROR();
   
    St = MatrixGPU(2 * Nline, 1, -1, 1);
    Sf = MatrixGPU(2 * Nline, 1, -1, 1);
  
    /*St.display(true);
    Sf.display(true);
    
    ZsRe.display();
    ZsIm.display();
    Yd.display();*/
    
    //CHECK_LAST_CUDA_ERROR();

   
#ifdef INSTRUMENTATION
    t2 = std::chrono::high_resolution_clock::now();
    timePerBlock.increment(0, 0, std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
    occurencePerBlock.increment(0, 0, 1);
#endif // INSTRUMENTATION
   //std::cout << " fin init" << std::endl;

}

bool GPUPFdistPQ::chekcase()
{
    bool transfertToDo = false;
    if (CoresLineBus.getPos()) {
        transfertToDo = true;
        CoresLineBus.transferCPU();
    }
    if (Nbus != (Nline + 1)) {
        std::cout << "wrong number of line "<< Nline << "against "<< Nbus << std::endl;
        return false;
    }
    for (int i = 0; i < Nline; i++) {
        if (CoresLineBus.get(i, 1) != (i + 1)) {
            std::cout << "wrong numerotation of line " << CoresLineBus.get(i, 1) << "against " << (i + 1) << std::endl;
            return false;
        }
        if (CoresLineBus.get(i, 0) > CoresLineBus.get(i, 1)) {
            std::cout << "wrong numeoration of bus " << CoresLineBus.get(i, 0) << "against " << CoresLineBus.get(i, 1) << std::endl;
            return false;
        }
    }
    if (ZsRe.getNLin() == 0 || Yd.getNLin() == 0 || ZsIm.getNLin() == 0) {
        std::cout << "matrice non defined, ZsRe, Zs Im, Yd" << std::endl;
        ZsRe.display();
        ZsIm.display();
        Yd.display();
        return false;
    }

    if (transfertToDo) {
        CoresLineBus.transferGPU();
    }
    //std::cout << "checkcase OK " << std::endl;

    return true;
}

void GPUPFdistPQ::updatePQ(MatrixGPU* PQ)
{
#ifdef INSTRUMENTATION
    t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION
    calculW0Bis(PQ);
#ifdef INSTRUMENTATION
    t2 = std::chrono::high_resolution_clock::now();
    timePerBlock.increment(0, 8, std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
    occurencePerBlock.increment(0, 8, 1);
#endif
}

void GPUPFdistPQ::solve()
{
    //std::cout << "solve distPQ" << std::endl;
    time = clock();
    err = 2 * epsPF;
    iter = 0;
    //std::cout << epsPF << " " << iterM << std::endl;
    status = 1;
    while (err > epsPF && iter < iterM) {
       
        calcS();
        
        //Jb.display();
        //VoltageRealImPre.display();

#ifdef INSTRUMENTATION
        t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION      
        calcVoltage();
        
#ifdef INSTRUMENTATION
        t2 = std::chrono::high_resolution_clock::now();
        timePerBlock.increment(0, 3, std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
        occurencePerBlock.increment(0, 3, 1);
        t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION
        

        err = VoltageRealIm.distance2(&VoltageRealImPre);
        
#ifdef INSTRUMENTATION
        t2 = std::chrono::high_resolution_clock::now();
        timePerBlock.increment(0, 7, std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
        occurencePerBlock.increment(0, 7, 1);
#endif // INSTRUMENTATION
        VoltageRealImPre.set(&VoltageRealIm);
       
        /*if (err < epsPF) {
            calcW();
            dW.subtract(&W0, &W); // dW = W0 - W
            err = dW.max2(); //err = ||dW||
        }*/

        iter++;
        //std::cout << err << " ";
       
    }
    //
#ifdef INSTRUMENTATION
    t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION
    calcW(true);
   // W.display(true);
#ifdef INSTRUMENTATION
    t2 = std::chrono::high_resolution_clock::now();
    timePerBlock.increment(0, 6, std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
    occurencePerBlock.increment(0, 6, 1);
#endif // INSTRUMENTATION
    /*std::cout << "tension bus entree puis sortie" << std::endl;
    W0.display(true);
    W.display(true);*/

    if (iter >= iterM) {
        status = 2;
        if (err > 100 * epsPF) {
            status = -1;
        }
        //std::cout << "fin solve " << iter << " " << err << std::endl;
    }

    time = clock() - time;


}

void GPUPFdistPQ::calcS()
{
    // step 2
#ifdef INSTRUMENTATION
    t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION
    //std::cout << "step 2" << std::endl;
    // Set receiving end branch ?ow equal to the sum of the demand at receiving end (s^k_d) and the power drawn in the admittance(y^k_d) connected to bus k
   
    calculStGPU << <numBlock, _blockSize >> > (St._matrixGPU, VoltageRealIm._matrixGPU, W0._matrixGPU, Yd._matrixGPU, Nbus);
    
#ifdef INSTRUMENTATION
    t2 = std::chrono::high_resolution_clock::now();
    timePerBlock.increment(0, 1, std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
    occurencePerBlock.increment(0, 1, 1);
    t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION
    //std::cout << "step 3" << std::endl;
    // step 3
    //Backward sweep: Perform current summation, starting from the branch with the biggest index and heading towards the branch
    //whose index is equal to 1. The current of branch k is added to the current of the branch whose index is equal to i = f(k)
    
    
    // : 117
    
    calculSGPU << <1, Nline, (Nline * (sizeof(bool) + sizeof(int)) + 2 * Nline * sizeof(float)) >> > (St._matrixGPU, Sf._matrixGPU, VoltageRealIm._matrixGPU, ZsRe._matrixGPU, ZsIm._matrixGPU, nChild._matrixGPU, Childs._matrixGPU, _indiceChildBegin._matrixGPU, Nbus);
   
    
#ifdef INSTRUMENTATION
    t2 = std::chrono::high_resolution_clock::now();
    timePerBlock.increment(0, 2, std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
    occurencePerBlock.increment(0, 2, 1);
#endif // INSTRUMENTATION
    //Jb.display();

}

void GPUPFdistPQ::calcW(bool end)
{
    
    calcWinterCar << <Nbus, _blockSize, B2 * sizeof(float) >> > (_Pintermediate._matrixGPU, _Qintermediate._matrixGPU, VoltageRealIm._matrixGPU, _Glin._matrixGPU, _Blin._matrixGPU, _CoresVoiLin._matrixGPU, _CoresBusLin._matrixGPU, _nLines._matrixGPU, Nbus);

    if (!end) { // pendant simu, la puissance à ce noeud est libre
        switch (_blockSize) {
        case 512:
            calcWGPU<512> << <Nbus, _blockSize >> > (W._matrixGPU, W0._matrixGPU, _Pintermediate._matrixGPU, _Qintermediate._matrixGPU, _CoresBusLin._matrixGPU, _nLines._matrixGPU, Nbus);
            break;
        case 256:
            calcWGPU<256> << <Nbus, _blockSize >> > (W._matrixGPU, W0._matrixGPU, _Pintermediate._matrixGPU, _Qintermediate._matrixGPU, _CoresBusLin._matrixGPU, _nLines._matrixGPU, Nbus);
            break;
        case 128:
            calcWGPU<128> << <Nbus, _blockSize >> > (W._matrixGPU, W0._matrixGPU, _Pintermediate._matrixGPU, _Qintermediate._matrixGPU, _CoresBusLin._matrixGPU, _nLines._matrixGPU, Nbus);
            break;
        case 64:
            calcWGPU< 64> << <Nbus, _blockSize >> > (W._matrixGPU, W0._matrixGPU, _Pintermediate._matrixGPU, _Qintermediate._matrixGPU, _CoresBusLin._matrixGPU, _nLines._matrixGPU, Nbus);
            break;
        case 32:
            calcWGPU< 32> << <Nbus, _blockSize >> > (W._matrixGPU, W0._matrixGPU, _Pintermediate._matrixGPU, _Qintermediate._matrixGPU, _CoresBusLin._matrixGPU, _nLines._matrixGPU, Nbus);
            break;
        case 16:
            calcWGPU< 16> << <Nbus, _blockSize >> > (W._matrixGPU, W0._matrixGPU, _Pintermediate._matrixGPU, _Qintermediate._matrixGPU, _CoresBusLin._matrixGPU, _nLines._matrixGPU, Nbus);
            break;
        case  8:
            calcWGPU<  8> << <Nbus, _blockSize >> > (W._matrixGPU, W0._matrixGPU, _Pintermediate._matrixGPU, _Qintermediate._matrixGPU, _CoresBusLin._matrixGPU, _nLines._matrixGPU, Nbus);
            break;
        case  4:
            calcWGPU<  4> << <Nbus, _blockSize >> > (W._matrixGPU, W0._matrixGPU, _Pintermediate._matrixGPU, _Qintermediate._matrixGPU, _CoresBusLin._matrixGPU, _nLines._matrixGPU, Nbus);
            break;
        case  2:
            calcWGPU<  2> << <Nbus, _blockSize >> > (W._matrixGPU, W0._matrixGPU, _Pintermediate._matrixGPU, _Qintermediate._matrixGPU, _CoresBusLin._matrixGPU, _nLines._matrixGPU, Nbus);
            break;
        case  1:
            calcWGPU<  1> << <Nbus, _blockSize >> > (W._matrixGPU, W0._matrixGPU, _Pintermediate._matrixGPU, _Qintermediate._matrixGPU, _CoresBusLin._matrixGPU, _nLines._matrixGPU, Nbus);
            break;
        }
    }
    else {
        switch (_blockSize) {
        case 512:
            calcWGPU<512> << <Nbus, _blockSize >> > (W._matrixGPU, _Pintermediate._matrixGPU, _Qintermediate._matrixGPU, _CoresBusLin._matrixGPU, _nLines._matrixGPU, Nbus);
            break;
        case 256:
            calcWGPU<256> << <Nbus, _blockSize >> > (W._matrixGPU, _Pintermediate._matrixGPU, _Qintermediate._matrixGPU, _CoresBusLin._matrixGPU, _nLines._matrixGPU, Nbus);
            break;
        case 128:
            calcWGPU<128> << <Nbus, _blockSize >> > (W._matrixGPU, _Pintermediate._matrixGPU, _Qintermediate._matrixGPU, _CoresBusLin._matrixGPU, _nLines._matrixGPU, Nbus);
            break;
        case 64:
            calcWGPU< 64> << <Nbus, _blockSize >> > (W._matrixGPU, _Pintermediate._matrixGPU, _Qintermediate._matrixGPU, _CoresBusLin._matrixGPU, _nLines._matrixGPU, Nbus);
            break;
        case 32:
            calcWGPU< 32> << <Nbus, _blockSize >> > (W._matrixGPU, _Pintermediate._matrixGPU, _Qintermediate._matrixGPU, _CoresBusLin._matrixGPU, _nLines._matrixGPU, Nbus);
            break;
        case 16:
            calcWGPU< 16> << <Nbus, _blockSize >> > (W._matrixGPU, _Pintermediate._matrixGPU, _Qintermediate._matrixGPU, _CoresBusLin._matrixGPU, _nLines._matrixGPU, Nbus);
            break;
        case  8:
            calcWGPU<  8> << <Nbus, _blockSize >> > (W._matrixGPU, _Pintermediate._matrixGPU, _Qintermediate._matrixGPU, _CoresBusLin._matrixGPU, _nLines._matrixGPU, Nbus);
            break;
        case  4:
            calcWGPU<  4> << <Nbus, _blockSize >> > (W._matrixGPU, _Pintermediate._matrixGPU, _Qintermediate._matrixGPU, _CoresBusLin._matrixGPU, _nLines._matrixGPU, Nbus);
            break;
        case  2:
            calcWGPU<  2> << <Nbus, _blockSize >> > (W._matrixGPU, _Pintermediate._matrixGPU, _Qintermediate._matrixGPU, _CoresBusLin._matrixGPU, _nLines._matrixGPU, Nbus);
            break;
        case  1:
            calcWGPU<  1> << <Nbus, _blockSize >> > (W._matrixGPU, _Pintermediate._matrixGPU, _Qintermediate._matrixGPU, _CoresBusLin._matrixGPU, _nLines._matrixGPU, Nbus);
            break;
        }

    }

}



int GPUPFdistPQ::calcVoltage()
{
   // std::cout << "step 4" << std::endl;
    //Forward sweep: The receiving end bus voltages are calculated with known branch currents and sending bus voltages. 
    //Sf.display(true);
   updateVoltage << <1, Nline, Nbus* (8*sizeof(bool) + 2 * sizeof(float)) >> > (VoltageRealIm._matrixGPU, ZsRe._matrixGPU, ZsIm._matrixGPU, Sf._matrixGPU, F._matrixGPU, Nbus, LastBus);
   
    return 0;
}


void GPUPFdistPQ::calcE()
{
    calcEGPU << <numBlock, _blockSize >> > (E._matrixGPU, VoltageRealIm._matrixGPU, Nbus);

}

MatrixGPU GPUPFdistPQ::getY()
{
    
    calcE();
    calculYGPU <<<numBlock, _blockSize >> > (Y._matrixGPU, E._matrixGPU, VoltageRealIm._matrixGPU, _Blin2._matrixGPU, _Glin2._matrixGPU, CoresLineBusGPU._matrixGPU, Nbus, Nline);
    
    return Y;
}

void GPUPFdistPQ::setE(MatrixGPU* Enew)
{
#ifdef INSTRUMENTATION
    t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION
    E = *Enew;
    if (!E.getPos()) {
        E.transferGPU();
    }
    initECar << <numBlock, _blockSize >> > (VoltageRealIm._matrixGPU, E._matrixGPU, Nbus);
    //CHECK_LAST_CUDA_ERROR();
    VoltageRealImPre.set(&VoltageRealIm);
    
#ifdef INSTRUMENTATION
    t2 = std::chrono::high_resolution_clock::now();
    timePerBlock.increment(0, 8, std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
    occurencePerBlock.increment(0, 8, 1);
#endif // INSTRUMENTATION
}

void GPUPFdistPQ::display2(bool all)
{
    std::cout.precision(3);
    float errV = err;
    W.transferCPU();
    W0.transferCPU();
    E.transferCPU();
    dW.transferCPU();

    if (iter == 0) {
        std::cout << "algorithm not launch" << std::endl;
        calcW(true);
       
        float temp = W.get(0, 0);
        float temp2 = W.get(Nbus, 0);
        W.set(0, 0, W0.get(0, 0));
        W.set(Nbus, 0, W0.get(Nbus, 0));
        dW.subtract(&W0, &W); // dW = W0 - W
        err = dW.max2(); //err = ||dW||
        W.set(0, 0, temp);
        W.set(Nbus, 0, temp2);
        
    }
    else if (iter < iterM) {
        std::cout << "method " << _name << " converged in " << iter << " iterations." << std::endl;
        std::cout << "Converged in " << (float)time / CLOCKS_PER_SEC << " seconds" << std::endl;
        
        std::cout << " Computation with float simple precision" << std::endl;
        float temp = W.get(0, 0);
       
        float temp2 = W.get(Nbus, 0);
        W.set(0, 0, W0.get(0, 0));
        W.set(Nbus, 0, W0.get(Nbus, 0));
        dW.subtract(&W0, &W); // dW = W0 - W
        err = dW.max2(); //err = ||dW||
        W.set(0, 0, temp);
        W.set(Nbus, 0, temp2);
        

    }
    else {
        std::cout << "method " << _name << " not converged in " << iter << " iterations." << std::endl;
        std::cout << "time taken " << (float)time / CLOCKS_PER_SEC << " seconds" << std::endl;
    }
    std::cout << "The power error of this state is " << err << std::endl;
    std::cout << "The tension error of this state is " << errV << std::endl;
    std::cout << "===============================================================|" << std::endl;
    std::cout << "      System Summary                                           |" << std::endl;
    std::cout << "===============================================================|" << std::endl;
    std::cout << "Buses            " << Nbus << std::endl;
    std::cout << "Branches         " << Nline << std::endl;
    std::cout << "Ploss            " << getPloss() << std::endl;
    std::cout << "Qloss            " << getQloss() << std::endl;


    std::cout << std::endl << std::endl;
    if (all) {

        std::cout << "===============================================================================================|" << std::endl;
        std::cout << "      Bus Data                                                                                 |" << std::endl;
        std::cout << "===============================================================================================|" << std::endl;
        std::cout << " Bus |          Voltage        |  Power = Generation  + Load   |  Init = Generation  + Load    |" << std::endl;
        std::cout << "  #  |    Mag(pu) |  Ang(deg)  |    P (pu)     |     Q (pu)    |    P (pu)     |     Q (pu)    |" << std::endl;
        std::cout << "-----|------------|------------|---------------|---------------|---------------|---------------|" << std::endl;

        //std::cout << 0 << "      " << E.get(Nbus, 0) << "             " << E.get(0, 0) * (abs(E.get(0, 0)) > 0.0001) * 180 / 3.1415 << "              " << (abs(W.get(0, 0)) > 0.0001) * W.get(0, 0) << "         " << (abs(W.get(Nbus, 0)) > 0.0001) * W.get(Nbus, 0) << std::endl;

        float seuil = 0.0001;
       
        
        std::cout << std::setw(5) << 0 << "|" << std::setw(11) << E.get(Nbus, 0) << "*|" << std::setw(11) << E.get(0, 0) * (abs(E.get(0, 0)) > seuil) * 180 / 3.1415
            << "*|" << std::setw(15) << (abs(W.get(0, 0)) > seuil) * W.get(0, 0) << "|" << std::setw(15) << (abs(W.get(Nbus, 0)) > seuil) * W.get(Nbus, 0)
            << "|" << std::setw(15) << W0.get(0, 0) << "|" << std::setw(15)
            << W0.get(Nbus, 0) << "|" << std::endl;
        for (int b = 1; b < Nbus; b++) {
            //std::cout.width(10);
            //std::cout << b << "      " << E.get(b + Nbus, 0) << "        " << E.get(b, 0) * (abs(E.get(b, 0)) > 0.0001) * 180 / 3.1415 << "          " << (abs(W.get(b, 0)) > 0.0001) * W.get(b, 0) << "         " << (abs(W.get(b + Nbus, 0)) > 0.0001) * W.get(b + Nbus, 0) << std::endl;
            std::cout << std::setw(5) << b << "|" << std::setw(11) << E.get(b + Nbus, 0) << " |" << std::setw(11)
                << E.get(b, 0) * (abs(E.get(b, 0)) > seuil) * 180 / 3.1415 << " |" << std::setw(15)
                << (abs(W.get(b, 0)) > seuil) * W.get(b, 0) << "|" << std::setw(15) << (abs(W.get(b + Nbus, 0)) > seuil) * W.get(b + Nbus, 0)
                << "|" << std::setw(15) << W0.get(b, 0) << "|" << std::setw(15)
                << W0.get(b + Nbus, 0) << "|" << std::endl;

        }
        
    }
    else {
        float seuil = 0.0001;
        std::cout << " Bus |          Voltage        |  Power = Generation  + Load   |  Init = Generation  + Load    |" << std::endl;
        std::cout << "  #  |    Mag(pu) |  Ang(deg)  |    P (pu)     |     Q (pu)    |    P (pu)     |     Q (pu)    |" << std::endl;
        std::cout << "-----|------------|------------|---------------|---------------|---------------|---------------|" << std::endl;
        
        
         std::cout << std::setw(5) << 0 << "|" << std::setw(11) << E.get(Nbus, 0) << "*|" << std::setw(11) << E.get(0, 0) * (abs(E.get(0, 0)) > seuil) * 180 / 3.1415
                << "*|" << std::setw(15) << (abs(W.get(0, 0)) > seuil) * W.get(0, 0) << "|" << std::setw(15) << (abs(W.get(Nbus, 0)) > seuil) * W.get(Nbus, 0)
                << "|" << std::setw(15) << W0.get(0, 0) << "|" << std::setw(15)
                << W0.get(Nbus, 0) << "|" << std::endl;
        
    }

    std::cout << "===============================================================================================|" << std::endl;
    std::cout << "                      END PRINT                                                                |" << std::endl;
    std::cout << "===============================================================================================|" << std::endl;
}



/*
for (int l = 0; l < Nline; l++) {
        int k = l + 1; // bus to
        float vRe = VoltageRealImPre.get(k, 0);
        float vIm = VoltageRealImPre.get(k + Nbus, 0);
        float p = -W0.get(k, 0);
        float q = -W0.get(k + Nbus, 0);
        float y = l > 0 ? Yd.get(l, 0) : 0;

        float SRe =  p;
        float SIm =  q - y * (vRe * vRe + vIm * vIm);
        St.set(l, 0, SRe);
        St.set(l + Nline, 0, SIm);
    }

*/
__global__ void calculStGPU(float* St, float* Voltage, float* W0, float* Yd, int B) {
    int thIdx = threadIdx.x + blockDim.x * blockIdx.x;
    int size = gridDim.x * blockDim.x;
    for (int k = thIdx + 1; k < B; k += size) { // bus to
        int l = k - 1; // line
        float vRe = Voltage[k];
        float vIm = Voltage[k + B];
        float p = -W0[k];
        float q = -W0[k + B];
        float y = Yd[l];

        float SRe = p;
        float SIm = q - y * (vRe * vRe + vIm * vIm);
        St[l] = SRe;
        St[l + B - 1] = SIm;

    }
}



/*
for (int k = 1; k < Nbus; k++) {
        // branch l entre le bus i=F(k) et k = l + 1;

        int i = F.get(k, 0); // busFrom
        int l = k - 1; // line


        float vRe = VoltageRealIm.get(i, 0);
        float vIm = VoltageRealIm.get(i  + Nbus, 0);
        float V2 = vRe * vRe + vIm * vIm;

        float zRe = ZsRe.get(l, 0);
        float zIm = ZsIm.get(l, 0);
        float SRe = Sf.get(l, 0);
        float SIm = Sf.get(l + Nline, 0);

        float vRe2 = vRe - (zRe * (SRe * vRe + SIm * vIm) + zIm * (SIm * vRe - SRe * vIm)) / V2;
        float vIm2 = vIm - (zIm * (SRe * vRe + SIm * vIm) - zRe * (SIm * vRe - SRe * vIm)) / V2;

        VoltageRealIm.set(k, 0, vRe2);
        VoltageRealIm.set(k + Nbus, 0, vIm2);
}
*/

__global__ void updateVoltage(float* Voltage, float* ZsRe, float* ZsIm, float* Sf, float* F, int B, int LastBus) {
    
    int line = threadIdx.x;
    int size = blockDim.x;
    // un seul block
    extern __shared__ float globalMemory[];
    float* VoltageSh = globalMemory;
    bool* hasFinished = (bool*) &globalMemory[2 * B];

    __shared__ bool notfinished;
    bool mustCompute = false;
    int L = B - 1;
    int bus = line + 1;
    if (line == 0) {
        notfinished = true;
    }
    for (int k = line; k < 2 * B; k += size) {
        VoltageSh[k] = Voltage[k];
    }
    __syncthreads();
    if (line < L) {
        hasFinished[line] = false;
        int busFrom = F[bus];
        float zRe = ZsRe[line];
        float zIm = ZsIm[line];
        float SRe = Sf[line];
        float SIm = Sf[line + L];
        mustCompute = (busFrom == 0);
    
        while (notfinished) {
            if (mustCompute) { // divergent mais on n'y peut rien
                float vRe = VoltageSh[busFrom];
                float vIm = VoltageSh[busFrom + B];
                float V2 = vRe * vRe + vIm * vIm;


                float vRe2 = vRe - (zRe * (SRe * vRe + SIm * vIm) + zIm * (SIm * vRe - SRe * vIm)) / V2;
                float vIm2 = vIm - (zIm * (SRe * vRe + SIm * vIm) - zRe * (SIm * vRe - SRe * vIm)) / V2;

                VoltageSh[bus] = vRe2;
                VoltageSh[bus + B] = vIm2;


                hasFinished[line] = true;
                if (line == LastBus - 1) {
                    notfinished = false;
                }
            }
            __syncthreads();
            // trouver qui doit tourner � la prochaine boucle
            mustCompute = !(hasFinished[line]) && hasFinished[busFrom - 1];
           /* if (line == 0) {
                notfinished = false;
            }
            __syncthreads();
                // tous ecrive la même chose
            if (!hasFinished[line]) {
                notfinished = true;
            }*/
            
                __syncthreads();
            }
        }
    __syncthreads();
    for (int k = line; k < 2 * B; k += size) {
        Voltage[k] = VoltageSh[k];
    }
}


/*

for (int l = Nline-1; l >= 0; l--) {
        int k = l + 1; // busTo
        int i = F.get(k, 0); // busFrom
        int lprev = i - 1;

        float SRe = St.get(l,0);
        float SIm = St.get(l + Nline, 0);
        float vRe = VoltageRealImPre.get(k, 0);
        float vIm = VoltageRealImPre.get(k + Nbus, 0);

        float SfRe = SRe + ZsRe.get(l, 0) * (SRe * SRe + SIm * SIm) / (vRe * vRe + vIm * vIm);
        float SfIm = SIm + ZsIm.get(l, 0) * (SRe * SRe + SIm * SIm) / (vRe * vRe + vIm * vIm);

        Sf.set(l, 0, SfRe);
        Sf.set(l + Nline, 0, SfIm);
        if (lprev > -1) {
            St.increment(lprev, 0, Sf.get(l, 0));
            St.increment(lprev + Nline, 0, Sf.get(l + Nline, 0));
        }

    }
  */



__global__ void calculSGPU(float* St, float* Sf, float* Voltage, float* ZsRe, float* ZsIm, float* nChild, float* Childs, float* indiceChildBegin, int B) {

    int L = B - 1;
    extern __shared__ float  globalMemory2[];
    float* SfSh        = (float*) globalMemory2;
    int*   ChildsSh    = (int*)  &globalMemory2[2 * L];
    bool*  hasfinished = (bool*) &ChildsSh[L];
   

  /**/


    __shared__ bool notfinished;

    int line = threadIdx.x;
    int step = blockDim.x;
    
    if (line == 0) {
        notfinished = true;
    }/**/
    for (int l = line; l < L; l += step) {
        hasfinished[l] = false;
        ChildsSh[l] = Childs[l];
        SfSh[l] = 0.0f;
        SfSh[l + L] = 0;
    } 

    __syncthreads();
   

     
   
   
    if (line < L) {
        //hasfinished[line] = false;
        int bus = line + 1;
        int indiceChild = (bus < (B - 1)) ? indiceChildBegin[bus] : 0;
        int nb = nChild[bus];
        bool mustCompute = (nb == 0);
        float vRe = Voltage[bus];
        float vIm = Voltage[bus + B];
        float vNorm = (vRe * vRe + vIm * vIm);
        float ZRe = ZsRe[line];
        float ZIm = ZsIm[line];
        float StRe = St[line];
        float StIm = St[line + L];
   
        __syncthreads();/**/
        while (notfinished) {
            if (mustCompute) { // divergent mais on n'y peut rien
                for (int i = 0; i < nb; i++) { // calcul St 
                    int c = ChildsSh[indiceChild + i];
                    int lineChild = c - 1;
                    StRe += SfSh[lineChild];
                    StIm += SfSh[lineChild + L];
                }
                St[line] = StRe;
                St[line + L] = StIm;

                // calcul Sf
           
                float StNorm = (StRe * StRe + StIm * StIm);

                float SfRe = StRe + ZRe * StNorm / vNorm;
                float SfIm = StIm + ZIm * StNorm / vNorm;

                SfSh[line] = SfRe;
                SfSh[line + L] = SfIm;
                
          
                hasfinished[line] = true;
                
                
                if (line == 0) {
                    notfinished = false;
                }
            }
            __syncthreads();
            // trouver qui doit tourner � la prochaine boucle
            mustCompute = !(hasfinished[line]);

            for (int i = 0; i < nb; i++) {
                int c = ChildsSh[indiceChild + i];
                int lineChild = c - 1;
                mustCompute = (mustCompute && hasfinished[lineChild]); // il suffit qu'un enfant n'a pas fini pour que cela soit false
            }
            __syncthreads();
        }
    }
   
    for (int l = line; l < L; l += step) {
        Sf[l] = SfSh[l];
        Sf[l + L] = SfSh[l + L];
    }

   

}




/*for (int b = 0; b < B2; b++) {
        Y.set(b, 0, E.get(b, 0));
    }
    int line = 0;

    for (int i = 0; i < Nbus; i++) {
        int k = CoresBusLin.get(i, 0);

        float ei = VoltageRealIm.get(i, 0);
        float fi = VoltageRealIm.get(Nbus + i, 0);

        for (int voisin = k + 1; voisin < (k + nLines.get(i, 0)); voisin++) {
            int j = CoresVoiLin.get(voisin, 0);
            float ej = VoltageRealIm.get(j, 0);
            float fj = VoltageRealIm.get(j + Nbus, 0);
            if (j > i) {
                float B = BgridLin.get(voisin, 0);
                float G = GgridLin.get(voisin, 0);
                float Pij = (ei * ei + fi * fi - ei * ej - fi * fj) * G + (ei * fj - ej * fi) * B;

                Y.set(2 * Nbus + line, 0, Pij);

                line++;
            }
        }
    }*/



