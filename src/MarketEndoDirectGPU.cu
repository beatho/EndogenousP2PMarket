#include "hip/hip_runtime.h"
#include "../head/MarketEndoDirectGPU.cuh"
#define MAX(X, Y) X * (X >= Y) + Y * (Y > X)

// get loss
// comuunication loss
// feasible point
// init 




MarketEndoDirectGPU::MarketEndoDirectGPU() : Method()
{
#if DEBUG_CONSTRUCTOR
	std::cout << " MarketEndoDirectGPU Constructor" << std::endl;
#endif // DEBUG_CONSTRUCTOR
	_name = NAME;
	timePerBlock = MatrixCPU(1, 9, 0); // Fb0, Fb1ab , Fb2, Fb3, Fb4, Fb5,FB6, Fb0'
	// si les sous ensemble ne sont pas accessible, tout est dans le premier.
	occurencePerBlock = MatrixCPU(1, 9, 0); //nb de fois utilis� pendant la simu
}


MarketEndoDirectGPU::MarketEndoDirectGPU(float rho) : Method()
{
#if DEBUG_CONSTRUCTOR
	std::cout << "default MarketEndoDirectGPU Constructor" << std::endl;
#endif // DEBUG_CONSTRUCTOR
	_name = NAME;
	_rho = rho;
	timePerBlock = MatrixCPU(1, 9, 0); // Fb0, Fb11, FB12, Fb2, Fb3, Fb4, Fb5, FB6, Fb0'
	// si les sous ensemble ne sont pas accessible, tout est dans le premier.
	occurencePerBlock = MatrixCPU(1, 9, 0); //nb de fois utilis� pendant la simu
}

MarketEndoDirectGPU::~MarketEndoDirectGPU()
{
	
}
void MarketEndoDirectGPU::setParam(float rho)
{
	_rho = rho;
}

bool MarketEndoDirectGPU::chekcase()
{
	if (_nBus != (_nLine + 1)) {
		std::cout << "wrong number of line " << _nLine << "against " << _nBus << std::endl;
		return false;
	}
	for (int i = 0; i < _nLine; i++) {
		if (CoresLineBusCPU.get(i, 1) != (i + 1)) {
			std::cout << "wrong numerotation of line " << CoresLineBusCPU.get(i, 1) << "against " << (i + 1) << std::endl;
			return false;
		}
		if (CoresLineBusCPU.get(i, 0) > CoresLineBusCPU.get(i, 1)) {
			std::cout << "wrong numeoration of bus " << CoresLineBusCPU.get(i, 0) << "against " << CoresLineBusCPU.get(i, 1) << std::endl;
			return false;
		}
	}
	if (ZsRe.getNLin() == 0  || ZsIm.getNLin() == 0) {
		std::cout << "matrice non defined, ZsRe, Zs Im, Yd" << std::endl;
		ZsRe.display();
		ZsIm.display();
		return false;
	}

	return true;
}

void MarketEndoDirectGPU::solve(Simparam* result, const Simparam& sim, const StudyCase& cas)
{
#ifdef DEBUG_SOLVE
	cas.display();
	sim.display(1);
#endif // DEBUG_SOLVE
	
	clock_t tall =clock();
#ifdef INSTRUMENTATION
	std::chrono::high_resolution_clock::time_point t1;
	std::chrono::high_resolution_clock::time_point t2;
#endif // INSTRUMENTATION

	// FB 0
	if (_id == 0) {
#ifdef INSTRUMENTATION
		t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION
		init(sim, cas);
		CHECK_LAST_CUDA_ERROR();
#ifdef INSTRUMENTATION
		hipDeviceSynchronize();
		t2 = std::chrono::high_resolution_clock::now();
		timePerBlock.increment(0, 0, std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
		occurencePerBlock.increment(0, 0, 1);
#endif // INSTRUMENTATION
	}
	
	_iterG = sim.getIterG();
	_nIterL = sim.getIterL();
	_stepG = sim.getStepG();
	
	
	float epsG = sim.getEpsG();
	_epsL = sim.getEpsL() / 20;

	_epsL *= _epsL;
	
	float fc = 0;
	float resG = 2 * epsG;
	_iterGlobal = 0;
	
	
	while (((_iterGlobal < _iterG) && (resG>epsG)) || (_iterGlobal <= _stepG)) {
		/*std::cout << "---------------------------------" << std::endl;
		std::cout << " X avant" << std::endl;
		X.display(true);
		
		//LAMBDALin.display();
		//Bt1.display();
		//TradeLin.display();	
		
		std::cout << " Q "   << std::endl;
		Q.display(true);
		std::cout << " Y "   << std::endl;
		Y.display(true);
		std::cout << " Mu "   << std::endl;
		Mu.display(true);
		std::cout << " Chat "   << std::endl;
		Chat.display(true);
		std::cout << " Bp2 " << std::endl;
		Bp2.display(true);
		std::cout << " P " << std::endl;
		P.display(true);*/
		//Pn.saveCSVForce("TestPnGPU2.csv", 11, 1);
		//X.saveCSVForce("TestXGPU2.csv", 11, 1);
		//Y.saveCSVForce("TestYGPU2.csv", 11, 1);
		//Chat.saveCSVForce("TestChatGPU2.csv", 11, 1);
#ifdef INSTRUMENTATION
		hipDeviceSynchronize();
		t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION
		updatePMarket(); // puissance et trade (ind�pendament du bus, m�me si on aurait pu r�soudre bus par bus)
		//std::cout << " P " << std::endl;
		//P.display(true);
		
		
#ifdef INSTRUMENTATION
		hipDeviceSynchronize();
		t2 = std::chrono::high_resolution_clock::now();
		timePerBlock.increment(0, 1, std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
		t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION

		//updateXWOCurrent(); // flux dans le r�seau, tension
		updateXWOCurrentCPU();
#ifdef INSTRUMENTATION
		hipDeviceSynchronize();
		t2 = std::chrono::high_resolution_clock::now();
		timePerBlock.increment(0, 2, std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
#endif // INSTRUMENTATION
		//Q.saveCSVForce("TestQGPU.csv", 11, 1);
		CommunicationX();
	
#ifdef INSTRUMENTATION
		hipDeviceSynchronize();
		t2 = std::chrono::high_resolution_clock::now();
		timePerBlock.increment(0, 3, std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
		t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION
		//Y.saveCSVForce("TestYGPU.csv", 11, 1);
		updateGlobalProb();
		
#ifdef INSTRUMENTATION
		hipDeviceSynchronize();
		t2 = std::chrono::high_resolution_clock::now();
		timePerBlock.increment(0, 4, std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
		t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION
		
		updateMu();
		
		updateChat();

#ifdef INSTRUMENTATION
		hipDeviceSynchronize();
		t2 = std::chrono::high_resolution_clock::now();
		timePerBlock.increment(0, 5, std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
#endif // INSTRUMENTATION
		// FB 4
		if (!(_iterGlobal % _stepG)) {
#ifdef INSTRUMENTATION
			t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION
			resG = updateRes(_iterGlobal / _stepG);
			//std::cout << _iterGlobal << " " << resF.get(0, _iterGlobal / _stepG) << " " << resF.get(1, _iterGlobal / _stepG) << " " << resF.get(2, _iterGlobal / _stepG) << std::endl;
			//resG = 1;
#ifdef INSTRUMENTATION
			hipDeviceSynchronize();
			t2 = std::chrono::high_resolution_clock::now();
			timePerBlock.increment(0, 6, std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
#endif // INSTRUMENTATION
		}
		//std::cout << iterGlobal << " " << _iterLocal << " " << resL << " " << resF.get(0, iterGlobal / stepG) << " " << resF.get(1, iterGlobal / stepG) << std::endl;
		_iterGlobal++;
	}
	//std::cout << _iterGlobal << " " << resF.get(0, (_iterGlobal - 1) / _stepG) << " " << resF.get(1, (_iterGlobal - 1) / _stepG) << " " << resF.get(2, (_iterGlobal - 1) / _stepG) << std::endl;


#ifdef INSTRUMENTATION	
	occurencePerBlock.increment(0, 1, _iterGlobal);
	occurencePerBlock.increment(0, 2, _iterGlobal);
	occurencePerBlock.increment(0, 3, _iterGlobal);
	occurencePerBlock.increment(0, 4, _iterGlobal);
	occurencePerBlock.increment(0, 5, _iterGlobal);
	occurencePerBlock.increment(0, 6, _iterGlobal / _stepG);

	t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION
	

	//setPnFromX << < _nBus, _blockSizeSmall >> > (Pn._matrixGPU, X._matrixGPU, _indiceBusBegin._matrixGPU, _CoresAgentBus._matrixGPU, _nAgentByBus._matrixGPU, _CoresAgentBusBegin._matrixGPU, _nAgent);


	MatrixCPU tradeLinCPU;
	TradeLin.toMatCPU(tradeLinCPU);
	MatrixCPU LAMBDALinCPU;
	LAMBDALin.toMatCPU(LAMBDALinCPU);
	MatrixCPU PnCPU;
	Pn.toMatCPU(PnCPU);
	

	int indice = 0;
	for (int idAgent = 0; idAgent < _nAgentTrue; idAgent++) {
		MatrixCPU omega(cas.getVoisin(idAgent));
		int Nvoisinmax = nVoisinCPU.get(idAgent, 0);
		for (int voisin = 0; voisin < Nvoisinmax; voisin++) {
			int idVoisin = omega.get(voisin, 0);
			trade.set(idAgent, idVoisin, tradeLinCPU.get(indice, 0));
			LAMBDA.set(idAgent, idVoisin, LAMBDALinCPU.get(indice, 0));
			indice = indice + 1;
		}
	}
	for (int idAgent = _nAgentTrue; idAgent < _nAgent; idAgent++) {
		for (int idVoisin = 0; idVoisin < _nAgentTrue; idVoisin++) {
			if (idVoisin != (idAgent - _nAgentTrue)) {
				trade.set(idAgent, idVoisin, tradeLinCPU.get(indice, 0));
				LAMBDA.set(idAgent, idVoisin, LAMBDALinCPU.get(indice, 0));
				indice = indice + 1;
			}

		}
	}
	
	
	/*PnCPU.set(0, 0, getPLoss());
	PnCPU.set(_nAgentTrue, 0, getQLoss());
	*/

	//Ct.display();
	//Tlocal.display();

	fc = calcFc(&Cost1, &Cost2, &Tlocal, &Pn, &Ct, &tempN1, &tempNN);
	// FB 5
	
	result->setResF(&resF);
	result->setIter(_iterGlobal);
	result->setPn(&PnCPU);
	result->setFc(fc);
	result->setLAMBDA(&LAMBDA);
	result->setTrade(&trade);

#ifdef INSTRUMENTATION
	hipDeviceSynchronize();  
	t2 = std::chrono::high_resolution_clock::now();
	timePerBlock.increment(0, 7, std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
	occurencePerBlock.increment(0, 7, 1);

	result->setTimeBloc(&timePerBlock, &occurencePerBlock);
#endif // INSTRUMENTATION
	tall = clock() - tall;
	timeMarketEndo = tall;
	//display();
	result->setTime((float)tall / CLOCKS_PER_SEC);
	
}

void MarketEndoDirectGPU::updateP0(const StudyCase& cas)
{
	_id = _id + 1;
#ifdef INSTRUMENTATION
	std::chrono::high_resolution_clock::time_point t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION
    
	Pmin = cas.getPmin();
	Pmax = cas.getPmax();
	Cost2 = cas.getb();
	Cp = cas.getb();


	MatrixCPU Lb(cas.getLb());
	MatrixCPU Ub(cas.getUb());
	matLb.transferCPU();
	matUb.transferCPU();

	int indice = 0;

	for (int idAgent = 0; idAgent < _nAgent; idAgent++) {
		int Nvoisinmax = nVoisinCPU.get(idAgent, 0);
		for (int voisin = 0; voisin < Nvoisinmax; voisin++) {
			matLb.set(indice, 0, Lb.get(idAgent, 0));
			matUb.set(indice, 0, Ub.get(idAgent, 0));
			indice = indice + 1;
		}
	}
	matLb.transferGPU();
	matUb.transferGPU();
	
	float Ploss = Pn.get(0, 0, false);
	float Qloss = Pn.get(_nAgentTrue, 0, false);
	// pour essayer que cela marche
	Pn.add(&Pmin, &Pmax);
	Pn.divide(2);

	// unleash powe
	Pmin.set(0, 0, -POWERLIMIT, true);
	Pmax.set(_nAgentTrue, 0, POWERLIMIT, true);
	Pmin.set(_nAgentTrue, 0, -POWERLIMIT, true);

	
	Pn.set(0, 0, Ploss, true);
	Pn.set(_nAgentTrue, 0, Qloss, true);

	Pb.set(0.0);
	Pbmax.set(0.0);
	Pbmin.set(0.0);
	ComputePFromAgentToBus();

	Pmin.divideT(&nVoisin);
	Pmax.divideT(&nVoisin);
	Cp.multiplyT(&nVoisin);

	initPQAgent << < _nBus, _blockSizeSmall >> > (X._matrixGPU, _indiceBusBegin._matrixGPU, _CoresAgentBus._matrixGPU, _nAgentByBus._matrixGPU, _CoresAgentBusBegin._matrixGPU, Pn._matrixGPU, _nAgent);
	CHECK_LAST_CUDA_ERROR();
	
	

	initDFSPQ << <1, _nBus, _nBus* (8*sizeof(bool) + sizeof(int)) >> > (X._matrixGPU, Pb._matrixGPU, nChild._matrixGPU, Childs._matrixGPU, _indiceBusBegin._matrixGPU, _indiceChildBegin._matrixGPU, _nBus);
	CHECK_LAST_CUDA_ERROR();
	
	computeLoss();
	CHECK_LAST_CUDA_ERROR();
	
	CommunicationX();
	CHECK_LAST_CUDA_ERROR();
	//Y.set(&X);
	
	updateChat();
	CHECK_LAST_CUDA_ERROR();
	
	//Cp.display(true);

#ifdef INSTRUMENTATION
	hipDeviceSynchronize(); 
	std::chrono::high_resolution_clock::time_point t2 = std::chrono::high_resolution_clock::now();
	timePerBlock.increment(0, 8, std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
	occurencePerBlock.increment(0, 8, 1);
#endif // INSTRUMENTATION

}

void MarketEndoDirectGPU::init(const Simparam& sim, const StudyCase& cas)
{
	 

	if (_CoresChatBegin.getPos()) {

		//_CoresChatBegin.transferCPU();
		//_indiceBusBegin.transferCPU();

		Ancestor.transferCPU();
		PosChild.transferCPU();

		_indiceChildBegin.transferCPU();
		Childs.transferCPU();

		ZsIm.transferCPU();
		ZsRe.transferCPU();
	}
	


	// intitilisation des matrixs et variables 
	
	clock_t t = clock();
	//std::cout << "init " << std::endl;
	_rho = sim.getRho();
	
	initMarket(sim, cas);
	 

	_iterG = sim.getIterG();
	_stepG = sim.getStepG();
	
	float epsG = sim.getEpsG();
	float epsGC = sim.getEpsGC();
	_ratioEps = epsG / epsGC;
	
	_nBus = cas.getNBus();
	_nBusWLoss = _nBus + 1;
	_nLine = cas.getNLine(true); // ne doit pas �tre r�duit ici !!!

	_debutloss =  3 * _nLine + 5 * _nBus + 2 * (_nAgentTrue - 1); // L = nChild.sum()
	_sizeEndoMarketTotal = _debutloss;
	_sizeChat = 4 * _nBus;
	
	if (losstype == LossType::CURRENT) {
		_sizeEndoMarketTotal += (_nBus + 2); // pertes et courants sauf premier bus ou + 2
	}
	else if (losstype == LossType::POWER) {
		_sizeEndoMarketTotal += _nAgent;
	}
	
	_numBlocksB = ceil((_nBus + _blockSize - 1) / _blockSize);
	_numBlocksH = ceil((_sizeEndoMarketTotal + _blockSize - 1) / _blockSize);
	
	
	_numLineByBlockY = _sizeEndoMarketTotal / 100 + 1; // on veut maimum de 100 blocks !!!


	//std::cout << _nAgentTrue << " " << _nBus << " " << _nLine << " " << _sizeEndoMarketTotal << std::endl;

	_CoresAgentBus = MatrixGPU(cas.getCoresAgentBusLin(), 1);
	_CoresAgentBusBegin = MatrixGPU(cas.getCoresAgentBusLinBegin(), 1);
	_nAgentByBus = MatrixGPU(cas.getNagentByBus(), 1);
	_nAgentByBusCPU = cas.getNagentByBus();
	PosAgent = MatrixGPU(_nAgentTrue, 1, 0, 1);

	 
	nChildCPU = MatrixCPU(_nBus, 1);

	CoresLineBusCPU = cas.getCoresLineBus(true);
	CoresLineBus = MatrixGPU(CoresLineBusCPU, 1);

	_CoresBusAgent = MatrixGPU(cas.getCoresBusAgentLin(), 1); // Cores[n] = b

	Ancestor = MatrixGPU(_nBus, 1, 0); // A_i = bus ant�c�dent de i
	PosChild = MatrixGPU(_nBus, 1, 0); // indice du bus i dans Child[Ai]
	Ancestor.set(0, 0, -1); // the slack bus has no ancestor
	
	ZsRe = cas.getZsRe();
	ZsIm = cas.getZsImag();
	ZsNorm = MatrixCPU(_nLine, 1);
	
	if (!chekcase()) {
		throw std::invalid_argument("not a radial case");
	}

	for (int lold = 0; lold < _nLine; lold++) {
		int l = lold + 1;
		int busTo = l ;
		int busFrom = CoresLineBusCPU.get(lold, 0);
		Ancestor.set(busTo, 0, busFrom);
		nChildCPU.increment(busFrom, 0, 1);
		ZsNorm.set(lold, 0, ZsRe.get(lold, 0) * ZsRe.get(lold, 0) + ZsIm.get(lold, 0) * ZsIm.get(lold, 0));
	}
	nChild = MatrixGPU(nChildCPU, 1);


	_rhoInv = 1 / _rho;
	resF = MatrixCPU(3, (_iterG / _stepG) + 1);

	
	
	
	//std::cout << " local resolution " << std::endl;
	// local resolution
	tempN2 = MatrixGPU(_nAgent, 1, 0, 1);
	tempB2 = MatrixGPU(2 * _nBus, 1, 0, 1);
	
	Pbmax = MatrixGPU(2 * _nBus, 1, 0, 1);
	Pbmin = MatrixGPU(2 * _nBus, 1, 0, 1);
	Pb = MatrixGPU(2 * _nBus, 1, 0, 1);
	CoresSoloBusAgent = MatrixGPU(_nBus, 1, -1, 1);
	
	_nAgentByBusCPU.increment(0, 0, -1);
	_nAgentOn0 = _nAgentByBusCPU.get(0, 0);
	
	removeLossAgent << <1, 1 >> > (_nAgentByBus._matrixGPU, _CoresAgentBusBegin._matrixGPU);
	initPosAgent << <_nBus, _blockSizeSmall >> > (PosAgent._matrixGPU, _nAgentByBus._matrixGPU, _CoresAgentBusBegin._matrixGPU, _CoresAgentBus._matrixGPU);

	//hipDeviceSynchronize();
	//CHECK_LAST_CUDA_ERROR();
	//Pb.saveCSVForce("TestPbGPU2.csv", 11, 1);
	ComputePFromAgentToBus();
	//Pb.saveCSVForce("TestPbGPU2.csv", 11, 1);
	//hipDeviceSynchronize();
	//CHECK_LAST_CUDA_ERROR();
	//_nAgentByBus.display();

	//std::cout << " creation " << std::endl;
	X = MatrixGPU(_sizeEndoMarketTotal, 1, 0, 1);
	Ypre = MatrixGPU(_sizeEndoMarketTotal, 1, 0, 1);
	Y = MatrixGPU(_sizeEndoMarketTotal, 1, 0, 1);
	Y.preallocateReduction();

	Mu = MatrixGPU(_sizeEndoMarketTotal, 1, 0, 1);
	
	tempN1 = MatrixGPU(_nAgent, 1, 0, 1);
	tempNN = MatrixGPU(_nTrade, 1, 0, 1);
	//tempM1 = new MatrixCPU[_nAgent];
	tempM = MatrixGPU(_sizeEndoMarketTotal, 1, 0, 1);
	
	sizeMarketEndoDirectGPU = MatrixGPU(_nBusWLoss, 1, 0, 1);
	sizeMarketEndoDirectGPU.preallocateReduction();
	sizeMarketEndoDirectGPUBig = MatrixGPU(_sizeEndoMarketTotal, 1, 0, 1);

	indiceBusBeginCPU = MatrixCPU(_nBusWLoss, 1);
	_indiceBusBeginBig = MatrixGPU(_sizeEndoMarketTotal, 1, 0, 1);
	CoresChatBeginCPU = MatrixCPU(_nBusWLoss, 1);

	int debut = 0;
	int debutChat = 0;
	for (int i = 0; i < _nBus; i++) {
		int m = nChildCPU.get(i, 0);
		int nB = _nAgentByBusCPU.get(i, 0);
		indiceBusBeginCPU.set(i, 0, debut);
		CoresChatBeginCPU.set(i, 0, debutChat);
		int sizeA = m * 3 + 5 + 2 * nB;
		debut += sizeA;
		debutChat += 4;
	}
	indiceBusBeginCPU.set(_nBus, 0, debut);
	CoresChatBeginCPU.set(_nBus, 0, debutChat);

	
	_CoresChatBegin = MatrixGPU(CoresChatBeginCPU, 1);
	_indiceBusBegin = MatrixGPU(indiceBusBeginCPU, 1);
	defineSizeBig << <_nBusWLoss, _blockSize >> > (sizeMarketEndoDirectGPUBig._matrixGPU, nChild._matrixGPU, _indiceBusBegin._matrixGPU, sizeMarketEndoDirectGPU._matrixGPU, _indiceBusBeginBig._matrixGPU, _nAgentByBus._matrixGPU, losstype, _nBus, _nAgentTrue);
	 
	
	_sizeEndoMarketMax = sizeMarketEndoDirectGPU.max2();
	Hinv = MatrixGPU(_sizeEndoMarketTotal, _sizeEndoMarketMax, 0, 1);
	Q = MatrixGPU(_sizeEndoMarketTotal, 1, 0, 1);
	
	Childs = MatrixGPU(_nLine, 1);
	PosChild = MatrixGPU(_nBus, 1, -1);

	Chat = MatrixGPU(_sizeChat, 1, 0, 1);


	MatrixGPU lowerBound(cas.getLowerBound(), 1); //voltage angle, voltage, line...
	MatrixGPU upperBound(cas.getUpperBound(), 1); //voltage angle, voltage, line...
	VoltageLimit = MatrixGPU( 2, _nBus, 0, 1); // min, max
	VoltageLimitReal = MatrixGPU( 2, _nBus, 0, 1); // min, max
	
	initVoltageBound << < _numBlocksB, _blockSize >> > (VoltageLimitReal._matrixGPU, VoltageLimit._matrixGPU, lowerBound._matrixGPU, upperBound._matrixGPU, nChild._matrixGPU, _nBus);
	VoltageLimit.toMatCPU(VoltageLimitCPU);

	MatrixCPU nChildTemp(_nBus, 1, 0);
	
	int debutChild = 0;
	
	_indiceChildBegin = MatrixGPU(_nBus, 1);
	
	for (int i = 0; i < _nBus; i++) {
		if (i > 0) {
			_indiceChildBegin.set(i - 1, 0, debutChild);

			int Ai = Ancestor.get(i, 0);
			Childs.set(_indiceChildBegin.get(Ai, 0) + nChildTemp.get(Ai, 0), 0, i);
			PosChild.set(i, 0, nChildTemp.get(Ai, 0));
			nChildTemp.increment(Ai, 0, 1);
			debutChild += nChildCPU.get(i - 1, 0);
		}

	}
	Ancestor.transferGPU();
	PosChild.transferGPU();
	
	//std::cout << " Hinv " << std::endl;
	debut = 0;
	for (int i = 0; i < _nBus; i++) {
		// (Pi, Qi, li, vi, pn..., qn..., vai, Pci ..., Qci... , lci...) !!!!!
		int m = nChildCPU.get(i, 0);
		int nB = _nAgentByBusCPU.get(i, 0);
		int sizeA = m * 3 + 5 + 2 * nB;
		MatrixCPU A(2 + 1 * (i > 0), sizeA);

		if (i > 0) {
			A.set(2, 0, 2 * ZsRe.get(i - 1, 0));
			A.set(2, 1, 2 * ZsIm.get(i - 1, 0));
			A.set(2, 2, -ZsNorm.get(i - 1, 0));
			A.set(2, 3, -1);
			A.set(2, 4 + 2 * nB, 1); // vai
			A.set(0, 0, -1);
			A.set(1, 1, -1);
		}
		for (int In = 0; In < nB; In++) {
			A.set(0, 4 + In, 1);
			A.set(1, 4 + nB + In, 1);
		}

		for (int j = 0; j < m; j++) {
			int c = Childs.get(_indiceChildBegin.get(i, 0) + j, 0);
			A.set(0, 5 + 2 * nB + j, 1); // Pci
			A.set(1, 5 + 2 * nB + m + j, 1); // Qci
			A.set(0, 5 + 2 * nB + 2 * m + j, -ZsRe.get(c - 1, 0)); // -R l
			A.set(1, 5 + 2 * nB + 2 * m + j, -ZsIm.get(c - 1, 0)); // -X l
		}

		//A.display();

		MatrixCPU temp33(2 + 1 * (i > 0), 2 + 1 * (i > 0));
		MatrixCPU temp3M(2 + 1 * (i > 0), sizeA);
		MatrixCPU tempMM(sizeA, sizeA);
		MatrixCPU tempMMbis(sizeA, sizeA);

		temp33.multiplyTrans(&A, &A);
		temp33.invertGaussJordan(&temp33);
		temp3M.MultiplyMatMat(&temp33, &A);
		tempMM.multiplyTrans(&A, &temp3M, 0);

		tempMMbis.setEyes(-1);
		tempMMbis.add(&tempMM);
		MatrixGPU tempMMGPU = MatrixGPU(tempMMbis, 1);

		Hinv.setBloc(debut, debut + sizeA, 0, sizeA, &tempMMGPU);
		debut += sizeA;
	}
	
	
	// bus factice
	float sumP = 0;
	float sumQ = 0;
	int sizeA = 0;
	MatrixGPU A;
	switch (losstype)
	{
	case LossType::POWER:
		sizeA = _nAgent;
		A = MatrixGPU(sizeA, sizeA);
		A.setEyes(-1);
		A.set(0, 0, 0);
		A.set(_nAgentTrue, _nAgentTrue, 0);
		for (int i = 1; i < _nAgentTrue; i++) {
			A.set(0, i, 1); // sum(p) + Ploss = 0
			A.set(_nAgentTrue, i + _nAgentTrue, 1); // Qloss + sum(q) = 0
		}
		A.transferGPU();
		Hinv.setBloc(debut, debut + sizeA, 0, sizeA, &A);
		/*for (int i = 0; i < _nAgentTrue; i++) {
			A[_nBus].set(0, i, 1); // sum(p) + Ploss = 0
			A[_nBus].set(1, i + _nAgentTrue, 1); // Qloss + sum(q) = 0
		}*/
		break;
	case LossType::CURRENT:
		sizeA = 2 + _nBus;
		A = MatrixGPU(sizeA, sizeA);
		A.setEyes(-1);
		A.set(0, 0, 0);
		A.set(1, 1, 0);
		for (int i = 0; i < _nLine; i++) {
			A.set(0, i + 3, ZsRe.get(i, 0)); // sum(p) + Ploss = 0
			A.set(1, i + 3, ZsIm.get(i, 0)); // Qloss + sum(q) = 0
		}
		A.transferGPU();
		Hinv.setBloc(debut, debut + sizeA, 0, sizeA, &A);
		//A[_nBus].set(0, 0, 1); // ploss
		//A[_nBus].set(1, 1, 1); // qloss
		break;
	}
	Hinv.divide(_rho);
	
	//Hinv.display(true);
	_indiceChildBegin.transferGPU();
	Childs.transferGPU();
	ZsIm.transferGPU();
	ZsRe.transferGPU();
	
	
	//std::cout << " init valeur " << std::endl;
	 
	initPQAgentV << < _nBus, _blockSizeSmall >> > (X._matrixGPU, _indiceBusBegin._matrixGPU, _CoresAgentBus._matrixGPU, _nAgentByBus._matrixGPU, _CoresAgentBusBegin._matrixGPU, Pn._matrixGPU, _nAgentTrue);
	 
	
	
	initDFSPQ << <1, _nBus, _nBus* (8*sizeof(bool) + sizeof(int)) >> > (X._matrixGPU, Pb._matrixGPU, nChild._matrixGPU, Childs._matrixGPU, _indiceBusBegin._matrixGPU, _indiceChildBegin._matrixGPU, _nBus);
	 
	communicateX << <_nBusWLoss, _blockSize >> > (X._matrixGPU, nChild._matrixGPU, Ancestor._matrixGPU, Childs._matrixGPU, _indiceBusBegin._matrixGPU, _indiceChildBegin._matrixGPU, _nAgentByBus._matrixGPU, _CoresBusAgent._matrixGPU, PosAgent._matrixGPU, losstype, _nBus, _nAgentTrue);
	 
	computeLoss();
	 
	
	/*
	X[_nBus].set(0, 0, getPLoss());
	X[_nBus].set(1, 0, getQLoss());*/
	
	Y.set(&X);

	
	//std::cout << "updateChat" << std::endl;
	updateChat();
	 
	
	/*std::cout << "--------" << std::endl;
	std::cout << " Pn " << std::endl;
	Pn.display();*/
	/*for (int i = 0; i < _nBus; i++) {
		std::cout << " X " << i << std::endl;
		X[i].display();
		std::cout << " Y " << i << std::endl;
		Y[i].display();
		std::cout << " Q " << i << std::endl;
		Q[i].display();
		std::cout << " Mu " << i << std::endl;
		Mu[i].display();
		std::cout << " Chat " << i << std::endl;
		Chat[i].display();
	}*/
	

	
	//std::cout << "rho " << _rhog << " rhoL " << _rhol << " rho1 " << _rho1 << std::endl;
	//std::cout << "fin init temps : " << (float)(clock() - t) / CLOCKS_PER_SEC << std::endl;
	//std::cout << "---------------------------------------------------------------------------------------" << std::endl;
}

void MarketEndoDirectGPU::initMarket(const Simparam& sim, const StudyCase& cas)
{
	if (CoresMatLin.getPos()) { // une copie en trop mais pour l'instant c'est ok...
		CoresMatLin.transferCPU();

		CoresLinAgent.transferCPU();
		CoresAgentLin.transferCPU();
		CoresLinVoisin.transferCPU();
		CoresLinTrans.transferCPU();

		Tlocal_pre.transferCPU();
		TradeLin.transferCPU();
		LAMBDALin.transferCPU();

		matLb.transferCPU();
		matUb.transferCPU();
		Ct.transferCPU();

		Pn.transferCPU();
		Pmin.transferCPU();
		Pmax.transferCPU();
	}


	_nAgentTrue = cas.getNagent();
	_nAgent = 2 * _nAgentTrue;
	
	if (_rhol == 0) {
		_rhol = _rho;
	}
	nVoisinCPU = cas.getNvoi();
	nVoisin = MatrixGPU(nVoisinCPU, 1);
	nVoisin.preallocateReduction();

	_nTrade = nVoisin.sum();
	_nTradeP = nVoisin.sum(0, _nAgentTrue);
	//std::cout << "nTrade " << _nTrade << " " << _nTradeP << std::endl;
	
	_nTradeQ = _nTrade - _nTradeP;
	if (_nTradeQ != (_nAgentTrue * (_nAgentTrue - 1))) {
		std::cout << "err MarketEndoDirectGPU : " << _nAgent << " " << _nAgentTrue << " " << _nTrade << " " << _nTradeP << " " << _nTradeQ << std::endl;
		throw std::invalid_argument("Agent must be fully conected for the Q echanges, WIP");
	}
	int nVoisinMax = nVoisin.max2();
	if (_blockSize * NMAXPEERPERTRHREAD < nVoisinMax) {
		std::cout << _blockSize << " " << NMAXPEERPERTRHREAD << " " << nVoisinMax << std::endl;
		throw std::invalid_argument("For this Method, an agent must not have more than 5120 peers");
	}
	_numBlocksN = ceil((_nAgent + _blockSize - 1) / _blockSize);
	_numBlocksM = ceil((_nTrade + _blockSize - 1) / _blockSize);
	
	
	
	
	if (initWithMarketClear) {
		ADMMMarketGPU market;
		Simparam res(sim);
		market.solve(&res, sim, cas);
		//res.display();
		LAMBDA = res.getLambda();
		trade = res.getTrade();
		Pn = res.getPn();
		//Pn.display();
		//std::cout << "****" << std::endl;
		
	}
	else {
		LAMBDA = sim.getLambda();
		trade = sim.getTrade();
		Pn = sim.getPn(); // somme des trades
	}
	Pmin = cas.getPmin();
	Pmax = cas.getPmax();
	 // unleash powe
	Pmin.set(0, 0, -POWERLIMIT);
	Pmax.set(_nAgentTrue, 0, POWERLIMIT);
	Pmin.set(_nAgentTrue, 0, -POWERLIMIT);
	
	Pn.transferGPU();
	Pn.preallocateReduction();
	Pmin.transferGPU();
	Pmax.transferGPU();

	if (Pn.max2() == 0) {
		Pn.add(&Pmin, &Pmax);
		Pn.divide(2);
		Pn.set(0, 0, 0, true);
	}
	/*Pn.display();
	LAMBDA.display();
	trade.display();*/
	 
	//CHECK_LAST_CUDA_ERROR();

	_at1 = _rho; // car that apparait 2 fois 
	_at2 = _rhol;

	MatrixCPU BETA(cas.getBeta());
	MatrixCPU Ub(cas.getUb());
	MatrixCPU Lb(cas.getLb());

	/*if (Ub.get(_nAgentTrue, 0) == 0) { // unleash power
		Ub.set(_nAgentTrue, 0, POWERLIMIT);
		Lb.set(_nAgentTrue, 0, -POWERLIMIT);
	}*/


	
	
	//std::cout << "mise sous forme lin�aire" << std::endl;
	


	CoresMatLin = MatrixGPU(_nAgent, _nAgentTrue, -1);
	CoresAgentLin = MatrixGPU(_nAgent + 1, 1);
	CoresLinAgent = MatrixGPU(_nTrade, 1);
	CoresLinVoisin = MatrixGPU(_nTrade, 1);
	CoresLinTrans = MatrixGPU(_nTrade, 1);

	Tlocal_pre = MatrixGPU(_nTrade, 1);
	TradeLin = MatrixGPU(_nTrade, 1);
	LAMBDALin = MatrixGPU(_nTrade, 1);

	matLb = MatrixGPU(_nTrade, 1);
	matUb = MatrixGPU(_nTrade, 1);
	Ct = MatrixGPU(_nTrade, 1);
	

	int indice = 0;
	//std::cout << " P " << std::endl;
	for (int idAgent = 0; idAgent < _nAgentTrue; idAgent++) { // P
		MatrixCPU omega(cas.getVoisin(idAgent));
		int Nvoisinmax = nVoisinCPU.get(idAgent, 0);
		for (int voisin = 0; voisin < Nvoisinmax; voisin++) {
			int idVoisin = omega.get(voisin, 0);
			matLb.set(indice, 0, Lb.get(idAgent, 0));
			matUb.set(indice, 0, Ub.get(idAgent, 0));
			Ct.set(indice, 0, BETA.get(idAgent, idVoisin));
			TradeLin.set(indice, 0, trade.get(idAgent, idVoisin));
			Tlocal_pre.set(indice, 0, trade.get(idAgent, idVoisin));
			LAMBDALin.set(indice, 0, LAMBDA.get(idAgent, idVoisin));
			CoresLinAgent.set(indice, 0, idAgent);
			CoresLinVoisin.set(indice, 0, idVoisin);
			CoresMatLin.set(idAgent, idVoisin, indice);
			indice = indice + 1;
		}
		CoresAgentLin.set(idAgent + 1, 0, indice);
	}
	//std::cout << " Q " << std::endl;
	for (int idAgent = _nAgentTrue; idAgent < _nAgent; idAgent++) { // Q
		for (int idVoisin = 0; idVoisin < _nAgentTrue; idVoisin++) {
			if (idVoisin != (idAgent - _nAgentTrue)) {
				matLb.set(indice, 0, Lb.get(idAgent, 0));
				matUb.set(indice, 0, Ub.get(idAgent, 0));
				TradeLin.set(indice, 0, trade.get(idAgent, idVoisin));
				Tlocal_pre.set(indice, 0, trade.get(idAgent, idVoisin));
				LAMBDALin.set(indice, 0, LAMBDA.get(idAgent, idVoisin));
				CoresLinAgent.set(indice, 0, idAgent);
				CoresLinVoisin.set(indice, 0, idVoisin + _nAgentTrue);
				CoresMatLin.set(idAgent, idVoisin, indice);
				indice = indice + 1;
			}
		}

		CoresAgentLin.set(idAgent + 1, 0, indice);
	}
	for (int lin = 0; lin < _nTrade; lin++) {
		int i = CoresLinAgent.get(lin, 0);
		int j = CoresLinVoisin.get(lin, 0);
		if (lin >= _nTradeP) {
			i -= _nAgentTrue;
		}

		int k = CoresMatLin.get(j, i);
		CoresLinTrans.set(lin, 0, k);
	}
	// transfert des mises lineaire
	matUb.transferGPU();
	matLb.transferGPU();
	Ct.transferGPU();

	Tlocal_pre.transferGPU();
	TradeLin.transferGPU();
	LAMBDALin.transferGPU();

	CoresAgentLin.transferGPU();
	CoresLinAgent.transferGPU();
	CoresLinVoisin.transferGPU();
	CoresMatLin.transferGPU();
	CoresLinTrans.transferGPU();

	//std::cout << "autres donn�e sur CPU" << std::endl;
	tempNN = MatrixGPU(_nTrade, 1, 0, 1);
	tempNN.preallocateReduction();
	tempN1 = MatrixGPU(_nAgent, 1, 0, 1); // plut�t que de re-allouer de la m�moire � chaque utilisation
	//MatrixCPU temp1N(1, _nAgent, 0, 1);

	/**/

	

	P = Pn; // moyenne des trades, ici c'est juste pour qu'il ait la m�me taille sans avoir besoin de se poser de question
	P.divideT(&nVoisin);
	Tlocal = MatrixGPU(_nTrade, 1, 0, 1);
	Tlocal.preallocateReduction();

	Cost1 = MatrixGPU(cas.geta(), 1);
	Cost2 = MatrixGPU(cas.getb(), 1);
	Ap1 = nVoisin;
	Ap2 = nVoisin;
	Ap3 = Cost1;	
	Ap123 = MatrixGPU(_nAgent, 1, 0, 1);

	
	
	Cp = Cost2;
	Bp1 = MatrixGPU(_nAgent, 1, 0, 1);
	Bp2 = MatrixGPU(_nAgent, 1, 0, 1);
	Bt1 = MatrixGPU(_nTrade, 1, 0, 1);
	Bt2 = MatrixGPU(_nTrade, 1, 0, 1);

	MU = MatrixGPU(sim.getMU(), 1); // facteur reduit i.e lambda_l/_rho
	Tmoy = P;


	Pmin.divideT(&nVoisin);
	Pmax.divideT(&nVoisin);

	/*std::cout << "Power bound" << std::endl;
	Pmin.display();
	Pmax.display();*/


	Ap1.multiply(_rhol);
	Ap2.multiply( _rho);
	Ap2.multiplyT(&nVoisin);
	Cp.multiplyT(&nVoisin);
	Tmoy.divideT(&nVoisin);

	Ap3.multiplyT(&nVoisin);
	Ap3.multiplyT(&nVoisin);
	Ap123.add(&Ap1, &Ap2);
	Ap123.add(&Ap3);

	//CoresLinTrans.display();
	//CoresAgentLin.display();
	/*std::cout << _at1 << " " << _at2 << std::endl;
	
	Ct.display(true);
	Ap1.display(true);
	Ap2.display(true);
	Ap3.display(true);
	Ap123.display(true);
	Cp.display(true);

	Pmin.display(true);
	Pmax.display(true);
	matLb.display(true);
	matUb.display(true);
	std::cout << "fin init market" << std::endl;*/
	
	//CHECK_LAST_CUDA_ERROR();
}




void MarketEndoDirectGPU::updateGlobalProb() {
	
	
	Ypre.swap(&Y);
	int numBlock = _sizeEndoMarketTotal/ _numLineByBlockY;
	switch (_blockSize) {
	case 512:
		updateY<512> << <numBlock, _blockSize >> > (Y._matrixGPU, Hinv._matrixGPU, Q._matrixGPU, sizeMarketEndoDirectGPUBig._matrixGPU, _indiceBusBeginBig._matrixGPU, _sizeEndoMarketMax, _sizeEndoMarketTotal);
		break;
	case 256:
		updateY<256> << <numBlock, _blockSize >> > (Y._matrixGPU, Hinv._matrixGPU, Q._matrixGPU, sizeMarketEndoDirectGPUBig._matrixGPU, _indiceBusBeginBig._matrixGPU, _sizeEndoMarketMax, _sizeEndoMarketTotal);
		break;
	case 128:
		updateY<128> << <numBlock, _blockSize >> > (Y._matrixGPU, Hinv._matrixGPU, Q._matrixGPU, sizeMarketEndoDirectGPUBig._matrixGPU, _indiceBusBeginBig._matrixGPU, _sizeEndoMarketMax, _sizeEndoMarketTotal);
		break;
	case 64:
		updateY< 64> << <numBlock, _blockSize >> > (Y._matrixGPU, Hinv._matrixGPU, Q._matrixGPU, sizeMarketEndoDirectGPUBig._matrixGPU, _indiceBusBeginBig._matrixGPU, _sizeEndoMarketMax, _sizeEndoMarketTotal);
		break;
	case 32:
		updateY< 32> << <numBlock, _blockSize >> > (Y._matrixGPU, Hinv._matrixGPU, Q._matrixGPU, sizeMarketEndoDirectGPUBig._matrixGPU, _indiceBusBeginBig._matrixGPU, _sizeEndoMarketMax, _sizeEndoMarketTotal);
		break;
	case 16:
		updateY< 16> << <numBlock, _blockSize >> > (Y._matrixGPU, Hinv._matrixGPU, Q._matrixGPU, sizeMarketEndoDirectGPUBig._matrixGPU, _indiceBusBeginBig._matrixGPU, _sizeEndoMarketMax, _sizeEndoMarketTotal);
		break;
	case  8:
		updateY<  8> << <numBlock, _blockSize >> > (Y._matrixGPU, Hinv._matrixGPU, Q._matrixGPU, sizeMarketEndoDirectGPUBig._matrixGPU, _indiceBusBeginBig._matrixGPU, _sizeEndoMarketMax, _sizeEndoMarketTotal);
		break;
	case  4:
		updateY<  4> << <numBlock, _blockSize >> > (Y._matrixGPU, Hinv._matrixGPU, Q._matrixGPU, sizeMarketEndoDirectGPUBig._matrixGPU, _indiceBusBeginBig._matrixGPU, _sizeEndoMarketMax, _sizeEndoMarketTotal);
		break;
	case  2:
		updateY<  2> << <numBlock, _blockSize >> > (Y._matrixGPU, Hinv._matrixGPU, Q._matrixGPU, sizeMarketEndoDirectGPUBig._matrixGPU, _indiceBusBeginBig._matrixGPU, _sizeEndoMarketMax, _sizeEndoMarketTotal);
		break;
	case  1:
		updateY<  1> << <numBlock, _blockSize >> > (Y._matrixGPU, Hinv._matrixGPU, Q._matrixGPU, sizeMarketEndoDirectGPUBig._matrixGPU, _indiceBusBeginBig._matrixGPU, _sizeEndoMarketMax, _sizeEndoMarketTotal);
		break;
	}

	Y.set(3, 0, 1, 1);
	Y.set(4 + 2 * _nAgentOn0, 0, 1, 1);


	// communication of y, mu

}

void MarketEndoDirectGPU::updateX()
{
	/*double x1, x2, x3, x4, c1, c2, c3, c4, lambdaLo, lambdaUp, delta, x3min, x3max, x4max, gamma, k2;
	double c1122;
	int nSol = 0;
	int typeSol = 0;
	int BestRoot = 0;
	double bestGamma = -1;
	double p = 0;
	int nRoot = 0;
	bool neg = false;

	for (int i = 0; i < _nBus; i++) {

		bool goodSol = false;
		k2 = sqrt(2.0 / (nChild.get(i, 0) + 1));
		typeSol = 0;
		if (i == 0) { // slack bus
			goodSol = true;
			c3 = -2 * Chat[i].get(2, 0) / k2;
			x1 = 0;
			x2 = 0;
			x4 = 0;
			x4max = 0;
			x3 = 1 / k2;
			gamma = 0;
			typeSol = 1;
		}
		else {
			c1 = -2 * Chat[i].get(0, 0);
			c2 = -2 * Chat[i].get(1, 0);
			c3 = -2 * Chat[i].get(2, 0) / k2;
			c4 = -2 * Chat[i].get(3, 0);
			c1122 = c1 * c1 + c2 * c2;


			x3min = VoltageLimit.get(i, 0);
			x3max = VoltageLimit.get(i, 1);
			x4max = FluxLimit.get(i - 1, 0);

			// case without constraint

			x1 = -c1 / 2;
			x2 = -c2 / 2;
			x3 = -c3 / 2;
			x4 = -c4 / 2;
			lambdaUp = 0;
			lambdaLo = 0;

			if (x3 < x3min) {
				x3 = x3min;
				lambdaLo = (2 * x3 + c3);
			}
			else if (x3 > x3max) {
				x3 = x3max;
				lambdaUp = -(2 * x3 + c3);
			}

			if (x4 > x4max) {
				x4 = x4max;
			}

			gamma = k2 * x4 - (x1 * x1 + x2 * x2) / x3; // ce n'est pas vraiment gamma, doit �tre positif
			//std::cout << "x 1 : " << x1 << " " << x2 << " " << x3 * k2 << " " << x4 << " " << (x1 * x1 + x2 * x2) / x3  - k2 * x4 << std::endl;

			if (gamma >= 0) {
				// the solution is good !
				typeSol = 1;
				goodSol = true;
			}
			else {
				if (c1122 == 0) { // cas d�g�n�r�
					std::cout << " bus " << i << " : c1= " << c1 << " c2=" << c2 << " c4=" << c4 << " gamma= " << gamma << std::endl;
					x4 = 0;
					goodSol = true;
				}
				else if (gamma > bestGamma) {
					typeSol = 1;
					bestGamma = gamma;
				}
			}
		}

		// cas x4 = x4 max 
		
			//x3 = x3max
		if (!goodSol) {
			x4 = x4max;
			x3 = x3max;
			p = sqrt((k2 * x4) / (c1122 * x3)); // plus ou mois ce truc !!!
			x1 = p * c1 * x3;
			x2 = p * c2 * x3;
			if (abs(c1) > 0) {
				gamma = -(2 * x1 + c1) * x3 / (2 * x1);
			}
			else {
				gamma = -(2 * x2 + c2) * x3 / (2 * x2);
			}
			lambdaUp = -(2 * x3 + c3 - gamma * (x1 * x1 + x2 * x2) / (x3 * x3));
			delta = k2 * gamma - 2 * x4 - c4;
			if (gamma >= 0 && lambdaUp >= 0 && delta > 0) {
				typeSol = 5;
				goodSol = true;
			}
			else if (gamma > bestGamma && lambdaUp > bestGamma && delta > bestGamma) {
				typeSol = 5;
				bestGamma = min(min(gamma, lambdaUp), delta);
			}
		}
		if (!goodSol) {
			p = -p;
			x1 = p * c1 * x3;
			x2 = p * c2 * x3;
			if (abs(c1) > 0) {
				gamma = -(2 * x1 + c1) * x3 / (2 * x1);
			}
			else {
				gamma = -(2 * x2 + c2) * x3 / (2 * x2);
			}
			lambdaUp = -(2 * x3 + c3 - gamma * (x1 * x1 + x2 * x2) / (x3 * x3));
			delta = k2 * gamma - 2 * x4 - c4;
			if (gamma >= 0 && lambdaUp >= 0 && delta >= 0) {
				// the solution is good 
				goodSol = true;
				typeSol = 5;
				//nSol = n;
			}
			if (gamma > bestGamma && lambdaUp > bestGamma && delta > bestGamma) {
				typeSol = 5;
				neg = true;
				bestGamma = min(min(gamma, lambdaUp), delta);
			}
		}
		//x3 = x3min
		if (!goodSol) {
			// cas x3 = xmin
			x3 = x3min;
			p = sqrt((k2 * x4) / (c1122 * x3));
			x1 = p * c1 * x3;
			x2 = p * c2 * x3;
			if (abs(c1) > 0) {
				gamma = -(2 * x1 + c1) * x3 / (2 * x1);
			}
			else {
				gamma = -(2 * x2 + c2) * x3 / (2 * x2);
			}

			lambdaLo = -(2 * x3 + c3 - gamma * (x1 * x1 + x2 * x2) / (x3 * x3));
			delta = k2 * gamma - 2 * x4 - c4;
			if (gamma >= 0 && lambdaLo >= 0 && delta >= 0) {
				// the solution is good 
				typeSol = 6;
				goodSol = true;
				//nSol = n;

			}
			else if (gamma > bestGamma && lambdaLo > bestGamma && delta > bestGamma) {
				typeSol = 6;
				bestGamma = min(min(gamma, lambdaLo), delta);
			}
		}
		if (!goodSol) {
			p = -p;
			x1 = p * c1 * x3;
			x2 = p * c2 * x3;
			if (abs(c1) > 0) {
				gamma = -(2 * x1 + c1) * x3 / (2 * x1);
			}
			else {
				gamma = -(2 * x2 + c2) * x3 / (2 * x2);
			}

			lambdaLo = -(2 * x3 + c3 - gamma * (x1 * x1 + x2 * x2) / (x3 * x3));
			delta = k2 * gamma - 2 * x4 - c4;
			if (gamma >= 0 && lambdaLo >= 0 && delta >= 0) {
				// the solution is good 
				goodSol = true;
				typeSol = 6;
				//nSol = n;

			}
			else if (gamma > bestGamma && lambdaLo > bestGamma && delta > bestGamma) {
				typeSol = 6;
				bestGamma = min(min(gamma, lambdaLo), delta);
				neg = true;
			}
		}
		// x3min <x3 < x3max
		if (!goodSol) {
			// cas tension libre
			coefPoly2[0] = (c3 + k2 * x4) / 2;
			coefPoly2[1] = sqrt(k2 * x4 * c1122) / 4;
			//std::cout << " polynome " << coefPoly2[0] << " " << coefPoly2[1] << std::endl;

			nRoot = resolveRealPolynome3without2term(root5, coefPoly2);
			for (int n = 0; n < nRoot; n++) {
				double sqrtX3 = root5[n];
				//std::cout << "root5 " << root5[n] << std::endl;
				if (sqrtX3 >= 0) {
					x3 = sqrtX3 * sqrtX3;
					p = sqrt((k2 * x4) / (c1122 * x3));

					x1 = p * c1 * x3;
					x2 = p * c2 * x3;


					if (abs(c1) > 0) {
						gamma = -(2 * x1 + c1) * x3 / (2 * x1);
					}
					else {
						gamma = -(2 * x2 + c2) * x3 / (2 * x2);
					}
					delta = k2 * gamma - 2 * x4 - c4;
					//std::cout << "x : " << x1 << " " << x2 << " " << x3 * k2 << " " << x4 << " " << gamma << " " << delta << std::endl;
					if (gamma >= 0 && delta >= 0 && x3 <= x3max && x3 >= x3min) {
						// the solution is good 
						typeSol = 7;
						goodSol = true;
						//nSol = n;
						break;
					}
					if (gamma > bestGamma && delta > bestGamma && (x3max - x3) > bestGamma && (x3 - x3min) > bestGamma) {
						typeSol = 7;
						bestGamma = min(min(min(gamma, (x3max - x3)), (x3 - x3min)), delta);
						BestRoot = n;
					}
				}

			}
		}
		if (!goodSol) {
			coefPoly2[0] = (c3 + k2 * x4) / 2;
			coefPoly2[1] = -sqrt(k2 * x4 * c1122) / 4;
			//std::cout << " polynome " << coefPoly2[0] << " " << coefPoly2[1] << std::endl;

			nRoot = resolveRealPolynome3without2term(root6, coefPoly2);
			for (int n = 0; n < nRoot; n++) {
				double sqrtX3 = root6[n];
				//std::cout << "root6 " << root6[n] << std::endl;
				if (sqrtX3 > 0) {
					x3 = sqrtX3 * sqrtX3;
					p = -sqrt((k2 * x4) / (c1122 * x3));

					x1 = p * c1 * x3;
					x2 = p * c2 * x3;

					if (abs(c1) > 0) {
						gamma = -(2 * x1 + c1) * x3 / (2 * x1);
					}
					else {
						gamma = -(2 * x2 + c2) * x3 / (2 * x2);
					}
					delta = k2 * gamma - 2 * x4 - c4;
					//std::cout << "x : " << x1 << " " << x2 << " " << x3 * k2 << " " << x4 << " " << gamma << " " << delta << std::endl;
					if (gamma >= 0 && delta >= 0 && x3 <= x3max && x3 >= x3min) {
						// the solution is good 
						typeSol = 8;
						goodSol = true;
						//nSol = n;
						break;
					}
					if (gamma > bestGamma && delta > bestGamma && (x3max - x3) > bestGamma && (x3 - x3min) > bestGamma) {
						typeSol = 8;
						bestGamma = min(min(min(gamma, (x3max - x3)), (x3 - x3min)), delta);
						BestRoot = n;
						neg = true;
					}
				}

			}

		}
		

		// cas x4 < x4 max 
			// case x3 = x3max lambdaLo = 0 delta = 0
		if (!goodSol) {
			x3 = x3max;

			coefPoly2[0] = 2 * (c4 / (k2 * x3) + 1);
			coefPoly2[1] = 1 / x3;
			coefPoly2[0] = coefPoly2[0] * k2 * k2 / (4 * c1122);
			coefPoly2[1] = coefPoly2[1] * k2 * k2 / (4 * c1122);

			nRoot = resolveRealPolynome3without2term(root2, coefPoly2);

			for (int n = 0; n < nRoot; n++) {
				p = root2[n];

				x1 = p * c1 * x3;
				x2 = p * c2 * x3;
				x4 = (x1 * x1 + x2 * x2) / (x3 * k2);
				gamma = (2 * x4 + c4) / k2;
				lambdaUp = -(2 * x3 + c3 - gamma * (x1 * x1 + x2 * x2) / (x3 * x3));
				//std::cout << "x2 : " << x1 << " " << x2 << " " << x3 * k2 << " " << x4 << " " << gamma << " " << lambdaUp << std::endl;
				if (gamma >= 0 && lambdaUp >= 0 && x4 <= x4max) {
					// the solution is good 
					goodSol = true;
					typeSol = 2;
					//nSol = n;
					break;
				}
				if (gamma > bestGamma && lambdaUp > bestGamma && (x4max - x4) > bestGamma) {
					typeSol = 2;
					bestGamma = min((x4max - x4), min(gamma, lambdaUp));
					BestRoot = n;
				}

			}
		}
			// case x3 = x3min lambdaUp = 0
		if (!goodSol) {
			x3 = x3min;

			coefPoly2[0] = 2 * (c4 / (k2 * x3) + 1);
			coefPoly2[1] = 1 / x3;
			coefPoly2[0] = coefPoly2[0] * k2 * k2 / (4 * c1122);
			coefPoly2[1] = coefPoly2[1] * k2 * k2 / (4 * c1122);

			nRoot = resolveRealPolynome3without2term(root3, coefPoly2);

			for (int n = 0; n < nRoot; n++) {
				p = root3[n];
				//std::cout << "poly " << coefPoly2[0] * p + coefPoly2[1] + p * p * p << std::endl;
				x1 = p * c1 * x3;
				x2 = p * c2 * x3;
				x4 = (x1 * x1 + x2 * x2) / (x3 * k2);
				gamma = (2 * x4 + c4) / k2;
				lambdaLo = (2 * x3 + c3 - gamma * (x1 * x1 + x2 * x2) / (x3 * x3));

				//std::cout << "x 3: " << x1 << " " << x2 << " " << x3 * k2 << " " << x4 << " " << gamma << " " << lambdaLo << std::endl;

				if (gamma >= 0 && lambdaLo >= 0 && x4 <= x4max) {
					// the solution is good !
					typeSol = 3;
					goodSol = true;
					break;
				}
				if (gamma > bestGamma && lambdaLo > bestGamma && (x4max - x4) > bestGamma) {
					typeSol = 3;
					bestGamma = min((x4max - x4), min(gamma, lambdaLo));
					BestRoot = n;
				}
			}
		}
		// case xmin<x3<xmax lambdaLo = 0 lambdaUp = 0
		if (!goodSol) {

			coefPoly3[0] = c1122 / k2 * (2 * c3 / k2 - c4);
			coefPoly3[1] = (c3 - 2 * c4 / k2);
			coefPoly3[2] = -1;
			coefPoly3[0] = coefPoly3[0] * k2 * k2 / (c1122 * c1122);
			coefPoly3[1] = coefPoly3[1] * k2 * k2 / (c1122 * c1122);
			coefPoly3[2] = coefPoly3[2] * k2 * k2 / (c1122 * c1122);

			nRoot = resvolveRealPolynome4without2term(root4, coefPoly3);

			for (int n = 0; n < nRoot; n++) {
				p = root4[n];
				//std::cout << "poly " <<p * p * p * p + coefPoly3[0] * p*p*p + coefPoly3[1]*p + coefPoly3[2] << std::endl;
				x3 = -(c1122 * p + 2 * c3) / (2 * (c1122 * p * p + 2));
				x1 = p * c1 * x3;
				x2 = p * c2 * x3;
				x4 = (x1 * x1 + x2 * x2) / (x3 * k2);
				gamma = (2 * x4 + c4) / k2;
				//std::cout << "x 4: " << x1 << " " << x2 << " " << x3 * k2 << " " << x4 << " " << gamma << " " << std::endl;

				if (gamma >= 0 && x3 <= x3max && x3 >= x3min && x4 <= x4max) {
					// the solution is good !
					typeSol = 4;
					goodSol = true;
					break;
				}if (gamma > bestGamma && (x3max - x3) > bestGamma && (x3 - x3min) > bestGamma && (x4max - x4) > bestGamma) {
					typeSol = 4;
					bestGamma = min((x4max - x4), min(min(gamma, (x3max - x3)), (x3 - x3min)));
					BestRoot = n;
				}
			}
		}
		

		
		if (!goodSol) {
			std::cout << "*|*" << bestGamma << " " << typeSol << std::endl;
			if (typeSol == 1) {
				// case without constraint
				x1 = -c1 / 2;
				x2 = -c2 / 2;
				x3 = -c3 / 2;
				x3 = (x3max - x3) * (x3 > x3max) + (x3min - x3) * (x3min > x3) + x3;
				x4 = -c4 / 2; // ou  (x1 * x1 + x2 * x2) / (k2 * x3)
				x4 = (x4max - x4) * (x4 > x4max) + x4;
			}
			else if (typeSol > 4) {
				x4 = x4max;
				if (typeSol == 5) {
					x3 = x3max;
				}
				else if (typeSol == 6) {
					x3 = x3min;
				}
				else if (typeSol == 7) {
					x3 = root5[BestRoot];
				}
				else if (typeSol == 8) {
					x3 = root6[BestRoot];
				}
				p = sqrt((k2 * x4) / (c1122 * x3));
				if (neg) {
					p = -p;
				}

				x1 = p * c1 * x3;
				x2 = p * c2 * x3;
			}
			else {
				if (typeSol == 2) {
					x3 = x3max;
					p = root2[BestRoot];
				}
				else if (typeSol == 3) {
					x3 = x3min;
					p = root3[BestRoot];
				}
				else if (typeSol == 4) {
					p = root4[BestRoot];
					x3 = -(c1122 * p + 2 * c3) / (2 * (c1122 * p * p + 2));
					x3 = (x3max - x3) * (x3 > x3max) + (x3min - x3) * (x3min > x3) + x3;
				}
				x1 = p * c1 * x3;
				x2 = p * c2 * x3;
				x4 = (x1 * x1 + x2 * x2) / (x3 * k2);
				x4 = (x4max - x4) * (x4 > x4max) + x4;
			}
		}

		// X =  {Pi, Qi, vi, li, vAi, (pn, qn), (Pci, Qci, lci) for all child Ci}


		if (typeSol) {
			if (x4 > x4max) {
				std::cout << "probleme bus " << i << " " << x4max << " " << goodSol << " " << typeSol << std::endl;
				std::cout << "x : " << x1 << " " << x2 << " " << x3 * k2 << " " << x4 << " " << std::endl;
			}

			X[i].set(0, 0, x1);
			X[i].set(1, 0, x2);
			X[i].set(2, 0, x3 * k2);
			X[i].set(3, 0, x4);
		}
		else {
			std::cout << "pas de solution, pas de changement" << std::endl;
			std::cout << "probleme bus " << i << " " << x4max << " " << goodSol << " " << typeSol << std::endl;
			std::cout << "x : " << x1 << " " << x2 << " " << x3 * k2 << " " << x4 << " " << std::endl;

		}
		
		
		
		//std::cout << "x F : " << x1 << " " << x2 << " " << x3*k2 << " " << x4 << " " << gamma << std::endl;
		
	}
	*/

}

void MarketEndoDirectGPU::updateXWOCurrent()
{
	updateXEndoMarket <<<_numBlocksB, _blockSize >> > (X._matrixGPU, Chat._matrixGPU, VoltageLimit._matrixGPU, nChild._matrixGPU, _CoresChatBegin._matrixGPU, _indiceBusBegin._matrixGPU, _nBus);
}

void MarketEndoDirectGPU::updateXWOCurrentCPU()
{

	X.transferCPU();
	Chat.transferCPU();
	double x1, x2, x3, x4, c1, c2, c3, c4, lambdaLo, lambdaUp, x3min, x3max, gamma, k2;
	double c1122;
	int nSol = 0;
	
	double p = 0;
	int nRoot = 0;

	for (int i = 1; i < _nBus; i++) {
		int typeSol = 0;
		int BestRoot = 0;
		double bestGamma = -1;
		bool goodSol = false;

		int begining = indiceBusBeginCPU.get(i, 0);
		int nC = nChildCPU.get(i, 0);
		int beginChat = CoresChatBeginCPU.get(i, 0);

		k2 = sqrt(2.0 / (nC + 1));
		
		c1 = -2 * Chat.get(beginChat, 0);
		c2 = -2 * Chat.get(beginChat + 1, 0);
		c4 = -2 * Chat.get(beginChat + 2, 0);
		c3 = -2 * Chat.get(beginChat + 3, 0) / k2;
		c1122 = c1 * c1 + c2 * c2;


		x3min = VoltageLimitCPU.get(0, i);
		x3max = VoltageLimitCPU.get(1, i);

		// case without constraint

		x1 = -c1 / 2;
		x2 = -c2 / 2;
		x3 = -c3 / 2;
		x4 = -c4 / 2;
		lambdaUp = 0;
		lambdaLo = 0;

		if (x3 < x3min) {
			x3 = x3min;
			lambdaLo = (2 * x3 + c3);
		}
		else if (x3 > x3max) {
			x3 = x3max;
			lambdaUp = -(2 * x3 + c3);
		}
		gamma = k2 * x4 - (x1 * x1 + x2 * x2) / x3; // ce n'est pas vraiment gamma, doit �tre positif
		//std::cout << "x 1 : " << x1 << " " << x2 << " " << x3 * k2 << " " << x4 << " " << (x1 * x1 + x2 * x2) / x3  - k2 * x4 << std::endl;

		if (gamma >= 0) {
			// the solution is good !
			goodSol = true;
		}
		else {
			if (gamma > bestGamma) {
				typeSol = 1;
				bestGamma = gamma;
			}
		}
		
		if (!goodSol) { // cas d�g�n�r�
			if (c1122 == 0) {
				//std::cout << " bus " << i << " : c1= " << c1 << " c2=" << c2 << " c4=" << c4 << " gamma= " << gamma << std::endl;

				x4 = 0;
				goodSol = true;
			}
		}
		// case x3 = x3max lambdaLo = 0
		if (!goodSol) {
			x3 = x3max;

			coefPoly2[0] = 2 * (c4 / (k2 * x3) + 1);
			coefPoly2[1] = 1 / x3;
			coefPoly2[0] = coefPoly2[0] * k2 * k2 / (4 * c1122);
			coefPoly2[1] = coefPoly2[1] * k2 * k2 / (4 * c1122);

			nRoot = resolveRealPolynome3without2term(root2, coefPoly2);

			for (int n = 0; n < nRoot; n++) {
				p = root2[n];

				x1 = p * c1 * x3;
				x2 = p * c2 * x3;
				x4 = (x1 * x1 + x2 * x2) / (x3 * k2);
				gamma = (2 * x4 + c4) / k2;
				lambdaUp = -(2 * x3 + c3 - gamma * (x1 * x1 + x2 * x2) / (x3 * x3));
				//std::cout << "x2 : " << x1 << " " << x2 << " " << x3 * k2 << " " << x4 << " " << gamma << " " << lambdaUp << std::endl;
				if (gamma >= 0 && lambdaUp >= 0) {
					// the solution is good 
					goodSol = true;
					//nSol = n;
					break;
				}
				if (gamma > bestGamma && lambdaUp > bestGamma) {
					typeSol = 2;
					bestGamma = min(gamma, lambdaUp);
					BestRoot = n;
				}

			}
			// case x3 = x3min lambdaUp = 0
			if (!goodSol) {
				x3 = x3min;

				coefPoly2[0] = 2 * (c4 / (k2 * x3) + 1);
				coefPoly2[1] = 1 / x3;
				coefPoly2[0] = coefPoly2[0] * k2 * k2 / (4 * c1122);
				coefPoly2[1] = coefPoly2[1] * k2 * k2 / (4 * c1122);

				nRoot = resolveRealPolynome3without2term(root3, coefPoly2);

				for (int n = 0; n < nRoot; n++) {
					p = root3[n];
					//std::cout << "poly " << coefPoly2[0] * p + coefPoly2[1] + p * p * p << std::endl;
					x1 = p * c1 * x3;
					x2 = p * c2 * x3;
					x4 = (x1 * x1 + x2 * x2) / (x3 * k2);
					gamma = (2 * x4 + c4) / k2;
					lambdaLo = (2 * x3 + c3 - gamma * (x1 * x1 + x2 * x2) / (x3 * x3));

					//std::cout << "x 3: " << x1 << " " << x2 << " " << x3 * k2 << " " << x4 << " " << gamma << " " << lambdaLo << std::endl;

					if (gamma >= 0 && lambdaLo >= 0) {
						// the solution is good !
						goodSol = true;
						break;
					}
					if (gamma > bestGamma && lambdaLo > bestGamma) {
						typeSol = 3;
						bestGamma = min(gamma, lambdaLo);
						BestRoot = n;
					}
				}
			}
			// case xmin<x3<xmax lambdaLo = 0 lambdaUp = 0
			if (!goodSol) {

				coefPoly3[0] = c1122 / k2 * (2 * c3 / k2 - c4);
				coefPoly3[1] = (c3 - 2 * c4 / k2);
				coefPoly3[2] = -1;
				coefPoly3[0] = coefPoly3[0] * k2 * k2 / (c1122 * c1122);
				coefPoly3[1] = coefPoly3[1] * k2 * k2 / (c1122 * c1122);
				coefPoly3[2] = coefPoly3[2] * k2 * k2 / (c1122 * c1122);

				nRoot = resvolveRealPolynome4without2term(root4, coefPoly3);

				for (int n = 0; n < nRoot; n++) {
					p = root4[n];
					//std::cout << "poly " <<p * p * p * p + coefPoly3[0] * p*p*p + coefPoly3[1]*p + coefPoly3[2] << std::endl;
					x3 = -(c1122 * p + 2 * c3) / (2 * (c1122 * p * p + 2));
					x1 = p * c1 * x3;
					x2 = p * c2 * x3;
					x4 = (x1 * x1 + x2 * x2) / (x3 * k2);
					gamma = (2 * x4 + c4) / k2;
					//std::cout << "x 4: " << x1 << " " << x2 << " " << x3 * k2 << " " << x4 << " " << gamma << " " << std::endl;

					if (gamma >= 0 && x3 <= x3max && x3 >= x3min) {
						// the solution is good !
						goodSol = true;
						break;
					}if (gamma > bestGamma && (x3max - x3) > bestGamma && (x3 - x3min) > bestGamma) {
						typeSol = 4;
						bestGamma = min(min(gamma, (x3max - x3)), (x3 - x3min));
						BestRoot = n;
					}
				}
			}
		}
		if (!goodSol) {
			if (typeSol == 1) {
				// case without constraint
				x1 = -c1 / 2;
				x2 = -c2 / 2;
				x3 = -c3 / 2;
				x3 = (x3max - x3) * (x3 > x3max) + (x3min - x3) * (x3min > x3) + x3;
				x4 = -c4 / 2; // ou  (x1 * x1 + x2 * x2) / (k2 * x3)
			}
			else {
				if (typeSol == 2) {
					x3 = x3max;
					p = root2[BestRoot];
				}
				else if (typeSol == 3) {
					x3 = x3min;
					p = root3[BestRoot];
				}
				else if (typeSol == 4) {
					p = root4[BestRoot];
					x3 = -(c1122 * p + 2 * c3) / (2 * (c1122 * p * p + 2));
					x3 = (x3max - x3) * (x3 > x3max) + (x3min - x3) * (x3min > x3) + x3;
				}
				x1 = p * c1 * x3;
				x2 = p * c2 * x3;
				x4 = (x1 * x1 + x2 * x2) / (x3 * k2);
			}
		}

		// X =  {Pi, Qi, vi, li, vAi, (pn, qn), (Pci, Qci, lci) for all child Ci}
		X.set(begining, 0, x1);
		X.set(begining + 1, 0, x2);
		X.set(begining + 2, 0, x4);
		X.set(begining + 3, 0, x3* k2);

		//std::cout << "x F : " << x1 << " " << x2 << " " << x3*k2 << " " << x4 << " " << gamma << std::endl;

	}
	X.transferGPU();
	Chat.transferGPU();

}

void MarketEndoDirectGPU::updatePMarket()
{
	/*std::cout << "objective loss P " << Y.get(17, 0, false) << " "<< Y.get(18, 0, false) << std::endl;
	std::cout << Ap1.get(1, 0) << " " << Ap2.get(1, 0) << " " << Ap3.get(1, 0) << " "  << Bp2.get(1, 0) << " " << Cp.get(1, 0) << std::endl;
	std::cout << _at1 << " " << _at2 << std::endl;
	Ct.display();
	
	std::cout << "Bt1" << std::endl;
	Bt1.display(true);*/
	
	if (getQLoss() > 0) {
		matUb.setBloc(_nTradeP, _nTradeP + _nAgentTrue - 1, 0, 1, _Qloss);
		matLb.setBloc(_nTradeP, _nTradeP + _nAgentTrue - 1, 0, 1, 0.0);
	}
	else {
		matLb.setBloc(_nTradeP, _nTradeP + _nAgentTrue - 1, 0, 1, _Qloss);
		matUb.setBloc(_nTradeP, _nTradeP + _nAgentTrue - 1, 0, 1, 0.0);
	}
	
	/*std::cout << "limit" << std::endl;
	matLb.display(true);
	matUb.display(true);*/

	
	updateLocalProb();
	/*std::cout << "result" << std::endl;
	Tlocal.display(true);
	P.display(true);*/
	
	
	TradeLin.swap(&Tlocal);
	/*std::cout << Ap1.get(0, 0) << " " << Bp1.get(0, 0) << std::endl;
	std::cout << " P" << std::endl;
	P.display();
	Tmoy.display();
	TradeLin.display();*/
	
	updateXPn<<<_nBusWLoss, _blockSizeSmall>>>(X._matrixGPU, Pn._matrixGPU, P._matrixGPU, nVoisin._matrixGPU, _indiceBusBegin._matrixGPU, _nAgentByBus._matrixGPU, _CoresAgentBusBegin._matrixGPU,  _CoresAgentBus._matrixGPU, losstype, _nAgentTrue, _nBus);
	
	
	
}

void MarketEndoDirectGPU::updateMu()
{
	updateMUGPU << <_numBlocksH, _blockSize >> > (Mu._matrixGPU, Y._matrixGPU, X._matrixGPU, _rho, _sizeEndoMarketTotal);

}



float MarketEndoDirectGPU::getPLoss()
{
	_Ploss =  Y.get(_debutloss, 0, false);
	
	return _Ploss;
}

float MarketEndoDirectGPU::getQLoss()
{
	int indice = 1;
	if (losstype == LossType::POWER) {
		indice = _nAgentTrue;
	}

	_Qloss = Y.get(_debutloss + indice, 0, false);
	
	return _Qloss;
}

void MarketEndoDirectGPU::computeLoss()
{

	int numBlock = 1;
	switch (_blockSize) {
	case 512:
		ComputeLoss<512> << <numBlock, _blockSize >> > (X._matrixGPU, Pn._matrixGPU, _indiceBusBegin._matrixGPU, ZsRe._matrixGPU, ZsIm._matrixGPU, losstype, _nAgentTrue, _nBus);
		break;
	case 256:
		ComputeLoss<256> << <numBlock, _blockSize >> > (X._matrixGPU, Pn._matrixGPU, _indiceBusBegin._matrixGPU, ZsRe._matrixGPU, ZsIm._matrixGPU, losstype, _nAgentTrue, _nBus);
		break;
	case 128:
		ComputeLoss<128> << <numBlock, _blockSize >> > (X._matrixGPU, Pn._matrixGPU, _indiceBusBegin._matrixGPU, ZsRe._matrixGPU, ZsIm._matrixGPU, losstype, _nAgentTrue, _nBus);
		break;
	case 64:
		ComputeLoss< 64> << <numBlock, _blockSize >> > (X._matrixGPU, Pn._matrixGPU, _indiceBusBegin._matrixGPU, ZsRe._matrixGPU, ZsIm._matrixGPU, losstype, _nAgentTrue, _nBus);
		break;
	case 32:
		ComputeLoss< 32> << <numBlock, _blockSize >> > (X._matrixGPU, Pn._matrixGPU, _indiceBusBegin._matrixGPU, ZsRe._matrixGPU, ZsIm._matrixGPU, losstype, _nAgentTrue, _nBus);
		break;
	case 16:
		ComputeLoss< 16> << <numBlock, _blockSize >> > (X._matrixGPU, Pn._matrixGPU, _indiceBusBegin._matrixGPU, ZsRe._matrixGPU, ZsIm._matrixGPU, losstype, _nAgentTrue, _nBus); 
		break;
	case  8:
		ComputeLoss<  8> << <numBlock, _blockSize >> > (X._matrixGPU, Pn._matrixGPU, _indiceBusBegin._matrixGPU, ZsRe._matrixGPU, ZsIm._matrixGPU, losstype, _nAgentTrue, _nBus);
		break;
	case  4:
		ComputeLoss<  4> << <numBlock, _blockSize >> > (X._matrixGPU, Pn._matrixGPU, _indiceBusBegin._matrixGPU, ZsRe._matrixGPU, ZsIm._matrixGPU, losstype, _nAgentTrue, _nBus);
		break;
	case  2:
		ComputeLoss<  2> << <numBlock, _blockSize >> > (X._matrixGPU, Pn._matrixGPU, _indiceBusBegin._matrixGPU, ZsRe._matrixGPU, ZsIm._matrixGPU, losstype, _nAgentTrue, _nBus);
		break;
	case  1:
		ComputeLoss<  1> << <numBlock, _blockSize >> > (X._matrixGPU, Pn._matrixGPU, _indiceBusBegin._matrixGPU, ZsRe._matrixGPU, ZsIm._matrixGPU, losstype, _nAgentTrue, _nBus);
		break;
	}
}



void MarketEndoDirectGPU::updateChat()
{
	int numBlock = _nBus;
	switch (_blockSizeSmall) {
	case 512:
		updateChatGPU3<512> << <numBlock, _blockSizeSmall >> > (Chat._matrixGPU, Y._matrixGPU, Mu._matrixGPU, nChild._matrixGPU, Ancestor._matrixGPU, PosChild._matrixGPU, Childs._matrixGPU, _indiceBusBegin._matrixGPU, _indiceChildBegin._matrixGPU, _CoresChatBegin._matrixGPU, _nAgentByBus._matrixGPU, _rho, losstype, _nBus);
		break;
	case 256:
		updateChatGPU3<256> << <numBlock, _blockSizeSmall >> > (Chat._matrixGPU, Y._matrixGPU, Mu._matrixGPU, nChild._matrixGPU, Ancestor._matrixGPU, PosChild._matrixGPU, Childs._matrixGPU, _indiceBusBegin._matrixGPU, _indiceChildBegin._matrixGPU, _CoresChatBegin._matrixGPU, _nAgentByBus._matrixGPU, _rho, losstype, _nBus);
		break;
	case 128:
		updateChatGPU3<128> << <numBlock, _blockSizeSmall >> > (Chat._matrixGPU, Y._matrixGPU, Mu._matrixGPU, nChild._matrixGPU, Ancestor._matrixGPU, PosChild._matrixGPU, Childs._matrixGPU, _indiceBusBegin._matrixGPU, _indiceChildBegin._matrixGPU, _CoresChatBegin._matrixGPU, _nAgentByBus._matrixGPU, _rho, losstype, _nBus);
		break;
	case 64:
		updateChatGPU3< 64> << <numBlock, _blockSizeSmall >> > (Chat._matrixGPU, Y._matrixGPU, Mu._matrixGPU, nChild._matrixGPU, Ancestor._matrixGPU, PosChild._matrixGPU, Childs._matrixGPU, _indiceBusBegin._matrixGPU, _indiceChildBegin._matrixGPU, _CoresChatBegin._matrixGPU, _nAgentByBus._matrixGPU, _rho, losstype, _nBus);
		break;
	case 32:
		updateChatGPU3< 32> << <numBlock, _blockSizeSmall >> > (Chat._matrixGPU, Y._matrixGPU, Mu._matrixGPU, nChild._matrixGPU, Ancestor._matrixGPU, PosChild._matrixGPU, Childs._matrixGPU, _indiceBusBegin._matrixGPU, _indiceChildBegin._matrixGPU, _CoresChatBegin._matrixGPU, _nAgentByBus._matrixGPU, _rho, losstype, _nBus);
		break;
	case 16:
		updateChatGPU3< 16> << <numBlock, _blockSizeSmall >> > (Chat._matrixGPU, Y._matrixGPU, Mu._matrixGPU, nChild._matrixGPU, Ancestor._matrixGPU, PosChild._matrixGPU, Childs._matrixGPU, _indiceBusBegin._matrixGPU, _indiceChildBegin._matrixGPU, _CoresChatBegin._matrixGPU, _nAgentByBus._matrixGPU, _rho, losstype, _nBus);
		break;
	case  8:
		updateChatGPU3<  8> << <numBlock, _blockSizeSmall >> > (Chat._matrixGPU, Y._matrixGPU, Mu._matrixGPU, nChild._matrixGPU, Ancestor._matrixGPU, PosChild._matrixGPU, Childs._matrixGPU, _indiceBusBegin._matrixGPU, _indiceChildBegin._matrixGPU, _CoresChatBegin._matrixGPU, _nAgentByBus._matrixGPU, _rho, losstype, _nBus);
		break;
	case  4:
		updateChatGPU3<  4> << <numBlock, _blockSizeSmall >> > (Chat._matrixGPU, Y._matrixGPU, Mu._matrixGPU, nChild._matrixGPU, Ancestor._matrixGPU, PosChild._matrixGPU, Childs._matrixGPU, _indiceBusBegin._matrixGPU, _indiceChildBegin._matrixGPU, _CoresChatBegin._matrixGPU, _nAgentByBus._matrixGPU, _rho, losstype, _nBus);
		break;
	case  2:
		updateChatGPU3<  2> << <numBlock, _blockSizeSmall >> > (Chat._matrixGPU, Y._matrixGPU, Mu._matrixGPU, nChild._matrixGPU, Ancestor._matrixGPU, PosChild._matrixGPU, Childs._matrixGPU, _indiceBusBegin._matrixGPU, _indiceChildBegin._matrixGPU, _CoresChatBegin._matrixGPU, _nAgentByBus._matrixGPU, _rho, losstype, _nBus);
		break;
	case  1:
		updateChatGPU3<  1> << <numBlock, _blockSizeSmall >> > (Chat._matrixGPU, Y._matrixGPU, Mu._matrixGPU, nChild._matrixGPU, Ancestor._matrixGPU, PosChild._matrixGPU, Childs._matrixGPU, _indiceBusBegin._matrixGPU, _indiceChildBegin._matrixGPU, _CoresChatBegin._matrixGPU, _nAgentByBus._matrixGPU, _rho, losstype, _nBus);
		break;
	}


	

	
	// pour puissance
	updateBp2();
	
	// pour �changes
	updateLAMBDABt1GPU << <_numBlocksM, _blockSize >> > (Bt1._matrixGPU, LAMBDALin._matrixGPU, TradeLin._matrixGPU, _rho, CoresLinTrans._matrixGPU, _nTrade);

	//Bt1.display();
}

void MarketEndoDirectGPU::CommunicationX()
{
/**/ // X = { Pi, Qi, li, vi,, (pn, qn), vAi (Pci, Qci, lci) for all child Ci }
	
	communicateX << <_nBusWLoss, _blockSize >> > (X._matrixGPU, nChild._matrixGPU, Ancestor._matrixGPU, Childs._matrixGPU, _indiceBusBegin._matrixGPU, _indiceChildBegin._matrixGPU, _nAgentByBus._matrixGPU, _CoresBusAgent._matrixGPU, PosAgent._matrixGPU, losstype, _nBus, _nAgentTrue);

	
	// Q udate in argmin 0.5yHy + Qy

	// Y = { Pi, Qi, vi, li, (pn ...), qn..., vAi,  Pci ... , Qci ... , lci ... for all child Ci }

	updateQ << <_numBlocksH, _blockSize >> > (Q._matrixGPU, X._matrixGPU, Mu._matrixGPU, _rho, _sizeEndoMarketTotal);


}


float MarketEndoDirectGPU::updateRes(int indice) 
{
	
	updateDiffGPU << <_numBlocksM, _blockSize >> > (tempNN._matrixGPU, Tlocal._matrixGPU, CoresLinTrans._matrixGPU, _nTrade);
	float resR = tempNN.max2();

	float resS = Tlocal.max2(&TradeLin); // nomalement * _rhog mais si _rhog est tres grand impossible que cela converge !!!




	float resSTemp = _rho *Y.max2(&Ypre); // 

	if (resSTemp > resS) {
		resS = resSTemp;
	}

	float resV = Y.max2(&X) * _ratioEps;
	
		

	resF.set(0, indice, resR);
	resF.set(1, indice, resS);
	resF.set(2, indice, resV);



	return MAX(MAX(resV, resS), resR);
}

float MarketEndoDirectGPU::updateResRhoFixe(int indice)
{
	updateDiffGPU << <_numBlocksM, _blockSize >> > (tempNN._matrixGPU, Tlocal._matrixGPU, CoresLinTrans._matrixGPU, _nTrade);
	float resR = tempNN.max2();

	float resS = Tlocal.max2(&TradeLin); // nomalement * _rhog mais si _rhog est tres grand impossible que cela converge !!!




	float resSTemp = _rho * Y.max2(&Ypre);

	if (resSTemp > resS) {
		resS = resSTemp;
	}

	float resV = Y.max2(&X);



	resF.set(0, indice, resR);
	resF.set(1, indice, resS);
	resF.set(2, indice, resV);


	return MAX(MAX(resV, resS), resR);
}

int MarketEndoDirectGPU::feasiblePoint()
{

	bool mustTrans = false;
	if (X.getPos()) {
		X.transferCPU();
		_indiceBusBegin.transferCPU();
		mustTrans = true;
	}
	// X  (Pi, Qi, li, vi, pn..., qn..., vai, Pci ..., Qci... , lci...) !!!!!

	MatrixCPU test(_nBus, 1, -1);
	int counter = 0;
	for (int bus = 0; bus < _nBus; bus++) {
		int begin = _indiceBusBegin.get(bus, 0);
		float Si = X.get(begin, 0) * X.get(begin, 0) + X.get(begin + 1, 0) * X.get(begin + 1, 0);
		float li = X.get(begin + 2, 0);
		float vi = X.get(begin + 3, 0);
		float err = Si - li * vi;
		test.set(bus, 0, err);
		if (abs(err) > 0.0001) {
			counter++;
		}
	}
	//std::cout << " erreur sur la relaXation " << test.max2() << " " << counter << std::endl;
	//test.display();

	if (mustTrans) {
		X.transferGPU();
		_indiceBusBegin.transferGPU();
	}
	resF.set(2, (_iterGlobal - 1) / _stepG, test.max2());
	return counter;
}

void MarketEndoDirectGPU::ComputePFromAgentToBus()
{
	int numBlock = _nBus;
	switch (_blockSizeSmall) {
	case 512:
		ComputePFromAgentToBusGPU<512> << <numBlock, _blockSizeSmall >> > (Pb._matrixGPU, Pbmin._matrixGPU, Pbmax._matrixGPU, CoresSoloBusAgent._matrixGPU, Pn._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, _CoresAgentBus._matrixGPU, _nAgentByBus._matrixGPU, _CoresAgentBusBegin._matrixGPU, _nAgentTrue, _nBus);
		break;
	case 256:
		ComputePFromAgentToBusGPU<256> << <numBlock, _blockSizeSmall >> > (Pb._matrixGPU, Pbmin._matrixGPU, Pbmax._matrixGPU, CoresSoloBusAgent._matrixGPU, Pn._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, _CoresAgentBus._matrixGPU, _nAgentByBus._matrixGPU, _CoresAgentBusBegin._matrixGPU, _nAgentTrue, _nBus);
		break;
	case 128:
		ComputePFromAgentToBusGPU<128> << <numBlock, _blockSizeSmall >> > (Pb._matrixGPU, Pbmin._matrixGPU, Pbmax._matrixGPU, CoresSoloBusAgent._matrixGPU, Pn._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, _CoresAgentBus._matrixGPU, _nAgentByBus._matrixGPU, _CoresAgentBusBegin._matrixGPU, _nAgentTrue, _nBus);
		break;
	case 64:
		ComputePFromAgentToBusGPU< 64> << <numBlock, _blockSizeSmall >> > (Pb._matrixGPU, Pbmin._matrixGPU, Pbmax._matrixGPU, CoresSoloBusAgent._matrixGPU, Pn._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, _CoresAgentBus._matrixGPU, _nAgentByBus._matrixGPU, _CoresAgentBusBegin._matrixGPU, _nAgentTrue, _nBus);
		break;
	case 32:
		ComputePFromAgentToBusGPU< 32> << <numBlock, _blockSizeSmall >> > (Pb._matrixGPU, Pbmin._matrixGPU, Pbmax._matrixGPU, CoresSoloBusAgent._matrixGPU, Pn._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, _CoresAgentBus._matrixGPU, _nAgentByBus._matrixGPU, _CoresAgentBusBegin._matrixGPU, _nAgentTrue, _nBus);
		break;
	case 16:
		ComputePFromAgentToBusGPU< 16> << <numBlock, _blockSizeSmall >> > (Pb._matrixGPU, Pbmin._matrixGPU, Pbmax._matrixGPU, CoresSoloBusAgent._matrixGPU, Pn._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, _CoresAgentBus._matrixGPU, _nAgentByBus._matrixGPU, _CoresAgentBusBegin._matrixGPU, _nAgentTrue, _nBus);
		break;
	case  8:
		ComputePFromAgentToBusGPU<  8> << <numBlock, _blockSizeSmall >> > (Pb._matrixGPU, Pbmin._matrixGPU, Pbmax._matrixGPU, CoresSoloBusAgent._matrixGPU, Pn._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, _CoresAgentBus._matrixGPU, _nAgentByBus._matrixGPU, _CoresAgentBusBegin._matrixGPU, _nAgentTrue, _nBus);
		break;
	case  4:
		ComputePFromAgentToBusGPU<  4> << <numBlock, _blockSizeSmall >> > (Pb._matrixGPU, Pbmin._matrixGPU, Pbmax._matrixGPU, CoresSoloBusAgent._matrixGPU, Pn._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, _CoresAgentBus._matrixGPU, _nAgentByBus._matrixGPU, _CoresAgentBusBegin._matrixGPU, _nAgentTrue, _nBus);
		break;
	case  2:
		ComputePFromAgentToBusGPU<  2> << <numBlock, _blockSizeSmall >> > (Pb._matrixGPU, Pbmin._matrixGPU, Pbmax._matrixGPU, CoresSoloBusAgent._matrixGPU, Pn._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, _CoresAgentBus._matrixGPU, _nAgentByBus._matrixGPU, _CoresAgentBusBegin._matrixGPU, _nAgentTrue, _nBus);
		break;
	case  1:
		ComputePFromAgentToBusGPU<  1> << <numBlock, _blockSizeSmall >> > (Pb._matrixGPU, Pbmin._matrixGPU, Pbmax._matrixGPU, CoresSoloBusAgent._matrixGPU, Pn._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, _CoresAgentBus._matrixGPU, _nAgentByBus._matrixGPU, _CoresAgentBusBegin._matrixGPU, _nAgentTrue, _nBus);
		break;
	}
}


// Market !!!!

void MarketEndoDirectGPU::updateLocalProb() {
	// FB 1a
	int numBlocks = _nAgent;
	switch (_blockSize) {
	case 512:
		updateTradePGPUSharedResidual<512> << <numBlocks, _blockSize >> > (Tlocal._matrixGPU, Tlocal_pre._matrixGPU, Tmoy._matrixGPU, P._matrixGPU, MU._matrixGPU, nVoisin._matrixGPU, _at1, _at2, Bt1._matrixGPU, Ct._matrixGPU,
			matLb._matrixGPU, matUb._matrixGPU, Ap1._matrixGPU, Ap2._matrixGPU, Ap123._matrixGPU, Bp2._matrixGPU, Cp._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, CoresAgentLin._matrixGPU, _epsL, _nIterL);
		break;
	case 256:
		updateTradePGPUSharedResidual<256> << <numBlocks, _blockSize >> > (Tlocal._matrixGPU, Tlocal_pre._matrixGPU, Tmoy._matrixGPU, P._matrixGPU, MU._matrixGPU, nVoisin._matrixGPU, _at1, _at2, Bt1._matrixGPU, Ct._matrixGPU,
			matLb._matrixGPU, matUb._matrixGPU, Ap1._matrixGPU, Ap2._matrixGPU, Ap123._matrixGPU, Bp2._matrixGPU, Cp._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, CoresAgentLin._matrixGPU, _epsL, _nIterL);
		break;
	case 128:
		updateTradePGPUSharedResidual<128> << <numBlocks, _blockSize >> > (Tlocal._matrixGPU, Tlocal_pre._matrixGPU, Tmoy._matrixGPU, P._matrixGPU, MU._matrixGPU, nVoisin._matrixGPU, _at1, _at2, Bt1._matrixGPU, Ct._matrixGPU,
			matLb._matrixGPU, matUb._matrixGPU, Ap1._matrixGPU, Ap2._matrixGPU, Ap123._matrixGPU, Bp2._matrixGPU, Cp._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, CoresAgentLin._matrixGPU, _epsL, _nIterL);
		break;
	case 64:
		updateTradePGPUSharedResidual< 64> << <numBlocks, _blockSize >> > (Tlocal._matrixGPU, Tlocal_pre._matrixGPU, Tmoy._matrixGPU, P._matrixGPU, MU._matrixGPU, nVoisin._matrixGPU, _at1, _at2, Bt1._matrixGPU, Ct._matrixGPU,
			matLb._matrixGPU, matUb._matrixGPU, Ap1._matrixGPU, Ap2._matrixGPU, Ap123._matrixGPU, Bp2._matrixGPU, Cp._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, CoresAgentLin._matrixGPU, _epsL, _nIterL);
		break;
	case 32:
		updateTradePGPUSharedResidual< 32> << <numBlocks, _blockSize >> > (Tlocal._matrixGPU, Tlocal_pre._matrixGPU, Tmoy._matrixGPU, P._matrixGPU, MU._matrixGPU, nVoisin._matrixGPU, _at1, _at2, Bt1._matrixGPU, Ct._matrixGPU,
			matLb._matrixGPU, matUb._matrixGPU, Ap1._matrixGPU, Ap2._matrixGPU, Ap123._matrixGPU, Bp2._matrixGPU, Cp._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, CoresAgentLin._matrixGPU, _epsL, _nIterL);
		break;
	case 16:
		updateTradePGPUSharedResidual< 16> << <numBlocks, _blockSize >> > (Tlocal._matrixGPU, Tlocal_pre._matrixGPU, Tmoy._matrixGPU, P._matrixGPU, MU._matrixGPU, nVoisin._matrixGPU, _at1, _at2, Bt1._matrixGPU, Ct._matrixGPU,
			matLb._matrixGPU, matUb._matrixGPU, Ap1._matrixGPU, Ap2._matrixGPU, Ap123._matrixGPU, Bp2._matrixGPU, Cp._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, CoresAgentLin._matrixGPU, _epsL, _nIterL);
		break;
	case  8:
		updateTradePGPUSharedResidual<  8> << <numBlocks, _blockSize >> > (Tlocal._matrixGPU, Tlocal_pre._matrixGPU, Tmoy._matrixGPU, P._matrixGPU, MU._matrixGPU, nVoisin._matrixGPU, _at1, _at2, Bt1._matrixGPU, Ct._matrixGPU,
			matLb._matrixGPU, matUb._matrixGPU, Ap1._matrixGPU, Ap2._matrixGPU, Ap123._matrixGPU, Bp2._matrixGPU, Cp._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, CoresAgentLin._matrixGPU, _epsL, _nIterL);
		break;
	case  4:
		updateTradePGPUSharedResidual<  4> << <numBlocks, _blockSize >> > (Tlocal._matrixGPU, Tlocal_pre._matrixGPU, Tmoy._matrixGPU, P._matrixGPU, MU._matrixGPU, nVoisin._matrixGPU, _at1, _at2, Bt1._matrixGPU, Ct._matrixGPU,
			matLb._matrixGPU, matUb._matrixGPU, Ap1._matrixGPU, Ap2._matrixGPU, Ap123._matrixGPU, Bp2._matrixGPU, Cp._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, CoresAgentLin._matrixGPU, _epsL, _nIterL);
		break;
	case  2:
		updateTradePGPUSharedResidual<  2> << <numBlocks, _blockSize >> > (Tlocal._matrixGPU, Tlocal_pre._matrixGPU, Tmoy._matrixGPU, P._matrixGPU, MU._matrixGPU, nVoisin._matrixGPU, _at1, _at2, Bt1._matrixGPU, Ct._matrixGPU,
			matLb._matrixGPU, matUb._matrixGPU, Ap1._matrixGPU, Ap2._matrixGPU, Ap123._matrixGPU, Bp2._matrixGPU, Cp._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, CoresAgentLin._matrixGPU, _epsL, _nIterL);
		break;
	case  1:
		updateTradePGPUSharedResidual<  1> << <numBlocks, _blockSize >> > (Tlocal._matrixGPU, Tlocal_pre._matrixGPU, Tmoy._matrixGPU, P._matrixGPU, MU._matrixGPU, nVoisin._matrixGPU, _at1, _at2, Bt1._matrixGPU, Ct._matrixGPU,
			matLb._matrixGPU, matUb._matrixGPU, Ap1._matrixGPU, Ap2._matrixGPU, Ap123._matrixGPU, Bp2._matrixGPU, Cp._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, CoresAgentLin._matrixGPU, _epsL, _nIterL);
		break;
	}
}


void MarketEndoDirectGPU::updateBp2()
{
	
	updateBp2GPU << <_nBusWLoss, _blockSizeSmall >> > (Bp2._matrixGPU, Y._matrixGPU, Mu._matrixGPU, _indiceBusBegin._matrixGPU, _CoresAgentBusBegin._matrixGPU, _CoresAgentBus._matrixGPU, _nAgentByBus._matrixGPU, losstype, _rho, _nBus, _nAgentTrue);

	Bp2.divideT(&nVoisin);

}

// autre

void MarketEndoDirectGPU::display() {

	X.transferCPU();
	Y.transferCPU();
	Mu.transferCPU();
	Pn.transferCPU();
	_indiceBusBegin.transferCPU();
	_nAgentByBus.transferCPU();
	VoltageLimitReal.transferCPU();
	Pmin.transferCPU();
	Pmax.transferCPU();
	Pbmax.transferCPU();
	Pbmin.transferCPU();
	Pb.set(0.0);
	Pb.transferCPU();

	Cost1.transferCPU();
	Cost2.transferCPU();
	for (int i = 0; i < _nBus; i++) {
		int Nb = _nAgentByBus.get(i, 0);
		int begin = _CoresAgentBusBegin.get(i, 0);
		for (int In = 0; In < Nb; In++) {
			int n = _CoresAgentBus.get(In + begin, 0);
			Pb.set(i, 0, Pb.get(i, 0) + Pn.get(n, 0));
			Pb.set(i + _nBus, 0, Pb.get(i + _nBus, 0) + Pn.get(n + _nAgentTrue, 0));
		}
	}



	if (_iterGlobal == 0) {
		std::cout << "algorithm not launch" << std::endl;
	}
	else if (_iterGlobal < _iterG) {
		std::cout << "method " << _name << " converged in " << _iterGlobal << " iterations." << std::endl;
		std::cout << "Converged in " << (float) timeMarketEndo / CLOCKS_PER_SEC << " seconds" << std::endl;

	}
	else {
		std::cout << "method " << _name << " not converged in " << _iterGlobal << " iterations." << std::endl;
		std::cout << "time taken " << (float) timeMarketEndo / CLOCKS_PER_SEC << " seconds" << std::endl;
	}
	std::cout << "The power error of this state is (constraint) " << resF.get(0, _iterGlobal / _stepG) << " and convergence " << resF.get(1, _iterGlobal / _stepG) << std::endl;
	std::cout << "===============================================================|" << std::endl;
	std::cout << "      System Summary                                           |" << std::endl;
	std::cout << "===============================================================|" << std::endl;
	std::cout << "Buses            " << _nBus << std::endl;
	std::cout << "Branches         " << _nLine << std::endl;
	std::cout << "Agent            " << _nAgentTrue << std::endl;
	std::cout << "Ploss            " << getPLoss() << std::endl;
	std::cout << "Qloss            " << getQLoss() << std::endl;


	std::cout << std::endl << std::endl;
	
	std::cout << "========================================================================================================|" << std::endl;
	std::cout << "      Bus Data                                                                                          |" << std::endl;
	std::cout << "========================================================================================================|" << std::endl;
	std::cout << " Bus |    Voltage  |   Power = Generation  + Load    |                Mu voltage and power              |" << std::endl;
	std::cout << "  #  |     Mag(pu) |    P (pu)      |      Q (pu)    |     V (pu)     |      P (pu)    |      Q (pu)    |" << std::endl;
	std::cout << "-----|-------------|----------------|----------------|----------------|----------------|----------------|" << std::endl;

		
	float seuil = 0.0001;
		
	for (int b = 0; b < _nBus; b++) {
		int begining = _indiceBusBegin.get(b, 0);
		std::cout << std::setw(5) << b << "|" << std::setw(12) << sqrt(X.get(begining + 3, 0)) << " |" << std::setw(16)
			<< Pb.get(b, 0) << "|" << std::setw(16) << Pb.get(b, 0)
			<< "|" << std::setw(16) << Mu.get(begining + 3, 0) << "|" << std::setw(16)
			<< Mu.get(begining, 0) << "|" << std::setw(16) << Mu.get(begining + 1, 0) << "|" << std::endl;

	}
	std::cout << std::endl << std::endl;
	std::cout << "========================================================================================================|" << std::endl;
	std::cout << "      Line Data                                                                                         |" << std::endl;
	std::cout << "========================================================================================================|" << std::endl;
	std::cout << " Line |    From     |    To      |                Upstream flow                      |    Constraint    |" << std::endl;
	std::cout << "  #   |    Bus      |    Bus     |   P (pu)   |   Q (pu)   |   l (pu)   |  Loss (pu) |      lmax        |" << std::endl;
	std::cout << "------|-------------|------------|------------|------------|------------|------------|------------------|" << std::endl;

	for (int l = 0; l < _nLine; l++) {
		int b = l + 1;
		int begining = _indiceBusBegin.get(b, 0);
		std::cout << std::setw(6) << l << "|" << std::setw(12) << CoresLineBus.get(l, 0) << " |" << std::setw(12)
			<< CoresLineBus.get(l, 1) << "|" << std::setw(16) << X.get(begining + 0, 0)
			<< "|" << std::setw(16) << X.get(begining + 1, 0) << "|" << std::setw(16)
			<< X.get(begining + 2, 0) << "|" << std::setw(19) << X.get(begining + 2, 0) * ZsRe.get(l, 0) << "|" << std::endl;
	}
	std::cout << std::endl << std::endl;
	std::cout << "========================================================================================================|" << std::endl;
	std::cout << "     Constraints                                                                                        |" << std::endl;
	std::cout << "========================================================================================================|" << std::endl;
	std::cout << " Bus | Voltage | Voltage | Voltage |        Power Injection          |          Power Injection         |" << std::endl;
	std::cout << "  #  | Mag(pu) | MIN(pu) |  MAX(pu)|  P (pu) | Pmin (pu) | Pmax (pu) |  Q (pu)  | Qmin (pu) | Qmax (pu) |" << std::endl;
	std::cout << "-----|---------|---------|---------|---------|-----------|-----------|----------|-----------|-----------|" << std::endl;
	
	for (int b = 0; b < _nBus; b++) {
		int begining = _indiceBusBegin.get(b, 0);
		std::cout << std::setw(5) << b << "|" << std::setw(8) << sqrt(Y.get(begining + 3, 0)) << " |" << std::setw(9)
			<< VoltageLimitReal.get(0, b) << "|" << std::setw(9) << VoltageLimitReal.get(1, b)
			<< "|" << std::setw(9) << Pb.get(b, 0) << "|" << std::setw(11)
			<< Pbmin.get(b, 0) << "|" << std::setw(11) << Pbmax.get(b, 0) << "|" << std::setw(10) << Pb.get(b + _nBus, 0)
			<< "|" << std::setw(11) << Pbmin.get(b + _nBus, 0) << "|" << std::setw(11) << Pbmax.get(b + _nBus, 0) << "|" << std::endl;
	}

	std::cout << std::endl << std::endl;
	std::cout << "========================================================================================================|" << std::endl;
	std::cout << "      Agent Data                                                                                        |" << std::endl;
	std::cout << "========================================================================================================|" << std::endl;
	std::cout << " Agent |  Bus  |  Cost   |  Cost   |        Power Injection          |          Power Injection         |" << std::endl;
	std::cout << "  #    |   #   |  a (pu) |  b (pu) |  P (pu) | Pmin (pu) | Pmax (pu) |  Q (pu)  | Qmin (pu) | Qmax (pu) |" << std::endl;
	std::cout << "-------|-------|---------|---------|---------|-----------|-----------|----------|-----------|-----------|" << std::endl;

	for (int n = 0; n < _nAgent; n++) {
		int b = _CoresBusAgent.get(n, 0);
		std::cout << std::setw(7) << n << "|" << std::setw(7) << b << "|" << std::setw(8) << Cost1.get(n, 0) << " |" << std::setw(9)
			<< Cost2.get(n, 0) << "|" << std::setw(9) << Pn.get(n, 0) << "|" << std::setw(11)
			<< Pmin.get(n, 0) << "|" << std::setw(11) << Pmax.get(n, 0) << "|" << std::setw(10) << Pn.get(n + _nAgent, 0)
			<< "|" << std::setw(11) << Pmin.get(n + _nAgent, 0) << "|" << std::setw(11) << Pmax.get(n + _nAgent, 0) << "|" << std::endl;
	}


	std::cout << "========================================================================================================|" << std::endl;
	std::cout << "                      END PRINT                                                                         |" << std::endl;
	std::cout << "========================================================================================================|" << std::endl;

}


template <unsigned int _blockSizeSmall>
__global__ void updateChatGPU3(float* Chat, float* Y, float* MU, float* nChild, float* Ancestor, float* posChild, float* Childs, float* indiceBusBegin, float* indiceChildBegin, float* CoresChatBegin, float* nAgentByBus, float _rho, int losstype, int nBus) {

	int bus = blockIdx.x;
	int index = threadIdx.x;
	int step = blockDim.x;
	
	int beginChat = CoresChatBegin[bus];

	__shared__ float shArr[_blockSizeSmall]; // c'est grand pour pas grand chose...


	int beginBus = indiceBusBegin[bus];
	int beginChild = (bus < (nBus - 1)) ? indiceChildBegin[bus] : 0;
	int childCount = nChild[bus];
	int Ai = Ancestor[bus];
	int nAgent = nAgentByBus[bus];
	int c = posChild[bus];
	float var = 0;
	int borne = 4;
	int divideVar = 2 + ((losstype == 1) && (index == 2)) * 1;
	int indiceLoss = indiceBusBegin[nBus];

	if (index < borne) {
		//float Phat, Qhat, lhat, vihat, pnhat..., qnhat...;
		var = Y[beginBus + index] / divideVar - MU[beginBus + index] / (divideVar * _rho);
		if (bus > 0) {
			if (index < 3) {
				int nAi = nChild[Ai];
				int nAgentAi = nAgentByBus[Ai];
				int indiceAncBus = indiceBusBegin[Ai] + 5 + 2 * nAgentAi + nAi * index + c;
				//var = indiceAncBus;
				var += Y[indiceAncBus] / divideVar - MU[indiceAncBus] / (divideVar * _rho);
				var += ((index == 2) && (losstype == 1)) ? (Y[indiceLoss + 2 + bus] / divideVar - MU[indiceLoss + 2 + bus] / (divideVar * _rho)) : 0;
			}
		}
	}
	float vhat = 0;
	float muhat = 0;
	for (int i = index; i < childCount; i += step) {
		int Bus2 = Childs[beginChild + i];
		int indiceBusChild = indiceBusBegin[Bus2];
		int nAgent2 = nAgentByBus[Bus2];
		muhat += MU[indiceBusChild + 4 + 2 * nAgent2]; // pas du tout coalescent
		vhat += Y[indiceBusChild + 4 + 2 * nAgent2]; // pas du tout coalescent
	}
	shArr[index] = vhat / (childCount + 1) - muhat / (_rho * (childCount + 1));
	__syncthreads();
	for (int size = _blockSizeSmall / 2; size > 0; size /= 2) { //uniform
		if (index < size) {
			shArr[index] += shArr[index + size];
		}
		__syncthreads();
	}

	if (index < borne) {
		if (index == 3) {
			var = shArr[0] + Y[beginBus + 3] / (childCount + 1) - MU[beginBus + 3] / (_rho * (childCount + 1)); //shArr[0];
		}
		Chat[beginChat + index] = var; // coalescent  !!!!
	}
}


/*
float phat, qhat;
	int divideP = 1;
	if (losstype == LossType::POWER) { // POWER
		divideP += 1;
	}

	for (int i = 0; i < _nBus; i++) {
		int Nb = _nAgentByBus.get(i, 0);
		int begin = _CoresAgentBusBegin.get(i, 0);
		for (int In = 0; In < Nb; In++) {
			int n = _CoresAgentBus.get(In + begin, 0);

			//std::cout << "bus " << i << " agent " << n << " en pos " << In << " Y " << Y[i].get(5 + 2 * In, 0) << " " <<
			//	Y[_nBus].get(n, 0) << " " << Mu[i].get(5 + 2 * In, 0) << " " << Mu[_nBus].get(n, 0);

			phat = Y[i].get(5 + 2 * In, 0) - Mu[i].get(5 + 2 * In, 0) / _rho;
			qhat = Y[i].get(6 + 2 * In, 0) - Mu[i].get(6 + 2 * In, 0) / _rho;
			if (losstype == LossType::POWER) {
				phat += Y[_nBus].get(n, 0) - Mu[_nBus].get(n, 0) / (_rho);
				qhat += Y[_nBus].get(n + _nAgentTrue, 0) - Mu[_nBus].get(n + _nAgentTrue, 0) / (_rho);
			}
			//std::cout <<  " phat " << phat/divideP << " Bp2 " << phat / (divideP * nVoisin.get(n,0)) << std::endl;
			//phat = (Y[i].get(5 + 2 * In, 0) + Y[_nBus].get(n, 0)) / 2			    - (Mu[i].get(5 + 2 * In, 0) + Mu[_nBus].get(n, 0)) / (2 * _rho);
			//qhat = (Y[i].get(6 + 2 * In, 0) + Y[_nBus].get(n + _nAgentTrue, 0)) / 2 - (Mu[i].get(6 + 2 * In, 0) + Mu[_nBus].get(n + _nAgentTrue, 0)) / (2 * _rho);


			Bp2.set(n, 0, phat / divideP);
			Bp2.set(n + _nAgentTrue, 0, qhat / divideP);
		}
	}
	// Y     (Ploss, Pn, Qloss Qn)

	float phatLoss = 0; // Y[_nBus].get(0, 0) - Mu[_nBus].get(0, 0) / _rho;
	float qhatLoss = 0; // Y[_nBus].get(1, 0) - Mu[_nBus].get(1, 0) / _rho;
	switch (losstype)
	{
	case LossType::POWER:
		phatLoss = Y[_nBus].get(0, 0) - Mu[_nBus].get(0, 0) / _rho;
		qhatLoss = Y[_nBus].get(_nAgentTrue, 0) - Mu[_nBus].get(_nAgentTrue, 0) / _rho;


		break;
	case LossType::CURRENT:
		phatLoss = Y[_nBus].get(0, 0) - Mu[_nBus].get(0, 0) / _rho;
		qhatLoss = Y[_nBus].get(1, 0) - Mu[_nBus].get(1, 0) / _rho;
		break;
	}


Bp2.set(0, 0, phatLoss);
Bp2.set(_nAgentTrue, 0, qhatLoss);

Bp2.divideT(&nVoisin);

*/

__global__ void updateBp2GPU(float* Bp2, float* Y, float* MU, float* indiceBusBegin, float* indiceAgentBegin, float* CoresAgentBus, float* nAgentByBus, int losstype, float rho, int nBus, int nAgent) {
	
	int bus	    = blockIdx.x;
	int thIdx   = threadIdx.x;
	int step	= blockDim.x;

	int begin   = indiceBusBegin[bus];
	int divideP = 1 + 1 * (losstype == 0);
	
	float phat = 0;
	float qhat = 0;

	if(bus < nBus) // bus normaux 
	{
		int Nb      = nAgentByBus[bus];
		int beginAgent = indiceAgentBegin[bus];
		int beginLoss = indiceBusBegin[nBus];
		
		for (int In = thIdx; In < Nb; In += step) {
			int n = CoresAgentBus[In + beginAgent];

			phat = Y[begin + 4 + In] - MU[begin + 4 + In] / rho;
			qhat = Y[begin + 4 + Nb + In] - MU[begin + 4 + Nb + In] / rho;
			
			if (losstype == 0) {
				phat += Y[beginLoss + n] - MU[beginLoss + n] / rho;
				qhat += Y[beginLoss + n + Nb] - MU[beginLoss + n + Nb] / rho;
			}
			

			Bp2[n] = phat / divideP;
			Bp2[n + nAgent] = qhat / divideP;
		}
	}
	else { // bus des pertes
		if (thIdx == 0) {
			if (losstype == 0) {
				phat = Y[begin] - MU[begin] / rho;
				qhat = Y[begin + nAgent] - MU[begin + nAgent] / rho;
			}
			else {
				phat = Y[begin] - MU[begin] / rho;
				qhat = Y[begin + 1] - MU[begin + 1] / rho;
			}
			Bp2[0] = phat;
			Bp2[nAgent] = qhat;
		}
	}

}

