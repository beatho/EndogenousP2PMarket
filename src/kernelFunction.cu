#include "hip/hip_runtime.h"
﻿
#include "../head/kernelFunction.cuh"


// OPF ADMM

__global__ void updateQ(float* Q, float* X, float* MU, float _rho, int sizeOPF) {

	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int step = blockDim.x * gridDim.x;


	for (int j = index; j < sizeOPF; j += step) {
		Q[j] = -(MU[j] + _rho * X[j]); //Q[i].set(j, 0, -(Mu[i].get(j, 0) + _rho * X[i].get(j, 0)));
	}
}


__global__ void updateMUGPU(float* Mu, float* Y, float* X, float rho, int sizeOPF) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int step = blockDim.x * gridDim.x;


	for (int j = index; j < sizeOPF; j += step) {
		Mu[j] = Mu[j] + rho * (X[j] - Y[j]);
	}
}

__global__ void removeLossAgent(float* _nAgentByBus, float* CoresAgentBusBegin) {

	int thIdx = threadIdx.x;
	if (thIdx == 0) {
		_nAgentByBus[0] = _nAgentByBus[0] - 1;
		CoresAgentBusBegin[0] = 1;
	}
}


__global__ void initVoltageBound(float* VlimReal, float* Vlim, float* constraintLo, float* constraintUp, float* nChild, int nBus) {

	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int step = blockDim.x * gridDim.x;

	for (int b = index; b < nBus; b += step) {
		float nb = sqrtf((nChild[b] + 1) / 2);
		float ub = constraintUp[b + nBus];
		float lb = constraintLo[b + nBus];

		VlimReal[b] = lb;
		VlimReal[b + nBus] = ub;

		Vlim[b] = lb * lb * nb;
		Vlim[b + nBus] = ub * ub * nb;
	}
}
__global__ void divideMultiplyByNagentByBus(float* Apt1, float* Apt2, float* PnTilde, float* PnTmin, float* PnTmax, float* nAgentByBus, float rhol, int nBus) {

	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int step = blockDim.x * gridDim.x;

	for (int b = index; b < nBus; b += step) {
		int nb = nAgentByBus[b];
		Apt1[b] = nb * rhol;
		Apt1[b + nBus] = nb * rhol;

		Apt2[b] = nb * nb * rhol;
		Apt2[b + nBus] = nb * nb * rhol;

		if (nb > 0) {
			PnTilde[b] = PnTilde[b] / nb;
			PnTilde[b + nBus] = PnTilde[b + nBus] / nb;

			PnTmin[b] = PnTmin[b] / nb;
			PnTmin[b + nBus] = PnTmin[b + nBus] / nb;

			PnTmax[b] = PnTmax[b] / nb;
			PnTmax[b + nBus] = PnTmax[b + nBus] / nb;
		}
	}
}


__global__ void initDFSPQ(float* X, float* Pb, float* nChild, float* Childs, float* indiceBusBegin, float* indiceChildBegin, int nBus) {


	extern __shared__ int globalMemory[];
	int* ChildsSh = globalMemory;
	bool* hasfinished = (bool*)(&(ChildsSh[nBus]));
	/*__shared__ int ChildsSh[2];
	__shared__ bool hasfinished[2];*/


	__shared__ bool notfinished;

	int bus = threadIdx.x; // one block and _blocksize = nBus !!!!

	if (bus == 0) {
		notfinished = true;
	}
	__syncthreads();

	if (bus < nBus) {
		hasfinished[bus] = false;
		int indice = indiceBusBegin[bus];
		int indiceChild = (bus < (nBus - 1)) ? indiceChildBegin[bus] : 0;
		int nb = nChild[bus];
		bool mustCompute = (nb == 0);
		ChildsSh[bus] = (bus < (nBus - 1)) ? Childs[bus] : 0;
		__syncthreads();
		while (notfinished) {
			__syncthreads();
			if (mustCompute) { // divergent mais on n'y peut rien
				float p = Pb[bus];
				float q = Pb[bus + nBus];
				for (int i = 0; i < nb; i++) {
					int c = ChildsSh[indiceChild + i];
					int indiceBusChild = indiceBusBegin[c];
					p += X[indiceBusChild];
					q += X[indiceBusChild + 1];
				}
				X[indice] = (bus > 0) * p;
				X[indice + 1] = (bus > 0) * q;
				float Si = p * p + q * q;
				X[indice + 2] = (bus > 0) * Si / X[indice + 3];
				hasfinished[bus] = true;
				if (bus == 0) {
					notfinished = false;
				}
				//notfinished = (bus != 0); // tous essaie d'ecrire la m�me chose sauf quand 0 sera tout seul �tant le premier noeud, le seul sans anc�tre
			}
			__syncthreads();
			// trouver qui doit tourner � la prochaine boucle
			mustCompute = !(hasfinished[bus]);

			for (int i = 0; i < nb; i++) {
				int c = ChildsSh[indiceChild + i];

				mustCompute = (mustCompute && hasfinished[c]); // il suffit qu'un enfant n'a pas fini pour que cela soit false
			}
			__syncthreads();
		}
	}

}
__global__ void initDFSPQ(float* X, float* nChild, float* Childs, float* indiceBusBegin, float* indiceChildBegin, int nBus) {


	extern __shared__ int globalMemory[];
	int* ChildsSh = globalMemory;
	bool* hasfinished = (bool*)(&(ChildsSh[nBus]));
	/*__shared__ int ChildsSh[2];
	__shared__ bool hasfinished[2];*/


	__shared__ bool notfinished;

	int bus = threadIdx.x; // one block and _blocksize = nBus !!!!

	if (bus == 0) {
		notfinished = true;
	}
	__syncthreads();

	if (bus < nBus) {
		hasfinished[bus] = false;
		int indice = indiceBusBegin[bus];
		int indiceChild = (bus < (nBus - 1)) ? indiceChildBegin[bus] : 0;
		int nb = nChild[bus];
		bool mustCompute = (nb == 0);
		ChildsSh[bus] = (bus < (nBus - 1)) ? Childs[bus] : 0;
		while (notfinished) {
			__syncthreads();
			if (mustCompute) { // divergent mais on n'y peut rien
				float p = X[indice + 4];
				float q = X[indice + 5];
				for (int i = 0; i < nb; i++) {
					int c = ChildsSh[indiceChild + i];
					int indiceBusChild = indiceBusBegin[c];
					p += X[indiceBusChild];
					q += X[indiceBusChild + 1];
				}
				X[indice] = (bus > 0) * p;
				X[indice + 1] = (bus > 0) * q;
				float Si = p * p + q * q;
				X[indice + 2] = (bus > 0) * Si / X[indice + 3];
				hasfinished[bus] = true;
				if (bus == 0) {
					notfinished = false;
				}
				//notfinished = (bus != 0); // tous essaie d'ecrire la m�me chose sauf quand 0 sera tout seul �tant le premier noeud, le seul sans anc�tre
			}
			__syncthreads();
			// trouver qui doit tourner � la prochaine boucle
			mustCompute = !(hasfinished[bus]);

			for (int i = 0; i < nb; i++) {
				int c = ChildsSh[indiceChild + i];
				mustCompute = (mustCompute && hasfinished[c]); // il suffit qu'un enfant n'a pas fini pour que cela soit false
			}
			__syncthreads();
		}
	}

}


__global__ void initPosAgent(float* PosAgent, float* nAgentByBus, float* CoresAgentBusBegin, float* CoresAgentBus) {

	int bus = blockIdx.x; // un bloc par bus
	int nB = nAgentByBus[bus];
	int index = threadIdx.x;
	int step = blockDim.x;
	int begin = CoresAgentBusBegin[bus];
	for (int i = index; i < nB; i += step) {
		int n = CoresAgentBus[begin + i];
		PosAgent[n] = i; // pas du tout coalescent
	}


}


__global__ void initPQAgentV(float* X, float* indiceBusBegin, float* CoresAgentBus, float* nAgentByBus, float* beginBusAgent, float* Pn, int nAgent) {
	int bus = blockIdx.x;
	int thI = threadIdx.x;
	int step = blockDim.x;
	int begin = beginBusAgent[bus];
	int nB = nAgentByBus[bus];
	int fin = begin + nB;
	int indiceBus = indiceBusBegin[bus];

	if (thI == 0) {
		X[indiceBus + 3] = 1; // vi
		X[indiceBus + 4 + 2 * nB] = 1; // vai
	}

	for (int i = begin + thI; i < fin; i += step) { // ecriture coalecente mais pas lecture
		int agent = CoresAgentBus[i];

		X[indiceBus + 4 + thI] = Pn[agent]; 
		X[indiceBus + 4 + nB + thI] = Pn[agent + nAgent]; 

	}
}
__global__ void initPQAgent(float* X, float* indiceBusBegin, float* CoresAgentBus, float* nAgentByBus, float* beginBusAgent, float* Pn, int nAgent) {
	int bus = blockIdx.x;
	int thI = threadIdx.x;
	int step = blockDim.x;
	int begin = beginBusAgent[bus];
	int nB = nAgentByBus[bus];
	int fin = begin + nB;
	int indiceBus = indiceBusBegin[bus];


	for (int i = begin + thI; i < fin; i += step) { // ecriture coalecente mais pas lecture
		int agent = CoresAgentBus[i];

		X[indiceBus + 4 + thI] = Pn[agent]; // pi = sum(pn)
		X[indiceBus + 4 + nB + thI] = Pn[agent + nAgent]; // qi = sum(pn)

	}
}

__global__ void initPQV(float* X, float* indiceBusBegin, float* nAgentByBus, float* PnTilde, int nBus) {
	int bus = blockIdx.x * blockDim.x + threadIdx.x;
	int step = blockDim.x * gridDim.x;

	for (int i = bus; i < nBus; i += step) { // pas coalescent
		int indiceBus = indiceBusBegin[i];
		X[indiceBus + 3] = 1; // vi

		// pi & qi
		int Nb = nAgentByBus[i];
		X[indiceBus + 4] = Nb * PnTilde[i]; // pi = sum(pn)
		X[indiceBus + 5] = Nb * PnTilde[i + nBus]; // qi = sum(pn)

	}
}
__global__ void initPQ(float* X, float* indiceBusBegin, float* nAgentByBus, float* PnTilde, int nBus) {
	int bus = blockIdx.x * blockDim.x + threadIdx.x;
	int step = blockDim.x * gridDim.x;

	for (int i = bus; i < nBus; i += step) { // pas coalescent
		int indiceBus = indiceBusBegin[i];

		// pi & qi
		int Nb = nAgentByBus[i];
		X[indiceBus + 4] = Nb * PnTilde[i]; // pi = sum(pn)
		X[indiceBus + 5] = Nb * PnTilde[i + nBus]; // qi = sum(pn)

	}
}


__global__ void defineSizeBig(float* sizeOPFADMMbig, float* nChild, float* CoresBusBegin, float* sizeOPFADMM, float* CoresBusBeginBig, float* nAgentByBus) {

	int bus = blockIdx.x; // un bloc par bus
	int thIdx = threadIdx.x;
	int step = blockDim.x;

	int nC = nChild[bus];
	int sizeOPF = nC * 3 + 5 + 2 * nAgentByBus[bus];
	int debut = CoresBusBegin[bus];

	if (thIdx == 0) {
		sizeOPFADMM[bus] = sizeOPF;
	}

	for (int i = thIdx; i < sizeOPF; i += step) {
		sizeOPFADMMbig[i + debut] = sizeOPF;
		CoresBusBeginBig[i + debut] = debut;
	}

}

__global__ void defineSizeBig(float* sizeOPFADMMbig, float* nChild, float* CoresBusBegin, float* sizeOPFADMM, float* CoresBusBeginBig, float* nAgentByBus, int lossType, int nBus, int nAgent) {

	int bus = blockIdx.x; // un bloc par bus
	int thIdx = threadIdx.x;
	int step = blockDim.x;

	if (bus < nBus) {
		int nC = nChild[bus];
		int sizeOPF = nC * 3 + 5 + 2 * nAgentByBus[bus];
		int debut = CoresBusBegin[bus];

		if (thIdx == 0) {
			sizeOPFADMM[bus] = sizeOPF;
		}

		for (int i = thIdx; i < sizeOPF; i += step) {
			sizeOPFADMMbig[i + debut] = sizeOPF;
			CoresBusBeginBig[i + debut] = debut;
		}
	}
	else { // loss
		if (lossType) { // Current
			int sizeOPF = nBus + 2;
			int debut = CoresBusBegin[bus];
			if (thIdx == 0) {
				sizeOPFADMM[bus] = sizeOPF;
			}

			for (int i = thIdx; i < sizeOPF; i += step) {
				sizeOPFADMMbig[i + debut] = sizeOPF;
				CoresBusBeginBig[i + debut] = debut;
			}
		}
		else { // POWER
			int sizeOPF = 2 * nAgent;
			int debut = CoresBusBegin[bus];
			if (thIdx == 0) {
				sizeOPFADMM[bus] = sizeOPF;
			}

			for (int i = thIdx; i < sizeOPF; i += step) {
				sizeOPFADMMbig[i + debut] = sizeOPF;
				CoresBusBeginBig[i + debut] = debut;
			}
		}

	}
	

}
__global__ void defineSizeBig(float* sizeOPFADMMbig, float* nChild, float* CoresBusBegin, float* sizeOPFADMM, float* CoresBusBeginBig) {

	int bus = blockIdx.x; // un bloc par bus
	int thIdx = threadIdx.x;
	int step = blockDim.x;

	int nC = nChild[bus];
	int sizeOPF = nC * 3 + 7;
	int debut = CoresBusBegin[bus];

	if (thIdx == 0) {
		sizeOPFADMM[bus] = sizeOPF;
	}

	for (int i = thIdx; i < sizeOPF; i += step) {
		sizeOPFADMMbig[i + debut] = sizeOPF;
		CoresBusBeginBig[i + debut] = debut;
	}

}


__global__ void updateXOPFADMM(float* X, float* Chat, float* Vbound, float* nAgentByBus, float* nChild, float* indiceBusBegin, float* CoresChatBegin,
	float* CoresAgentBusBegin, float* CoresAgentBus, float* Cost1, float* Cost2, float* Pmin, float* Pmax, float rho, int nBus, int nAgent, bool Lagrange) {

	int bus = blockIdx.x;
	int index = threadIdx.x;
	int step = blockDim.x;
	int beginChat = CoresChatBegin[bus];
	double x1, x2, x3, x4, c1, c2, c3, c4, lambdaLo, lambdaUp, x3min, x3max, gamma, k2; // double peut �tre necessaire
	double c1122; // c3 : voltage -> indice + 3, c4 : current -> indice + 2;
	double coefPoly2[2];
	double root2[4];
	double root3[4];
	double root4[4];
	double coefPoly3[3];
	int typeSol = 0;
	int BestRoot = 0;
	double bestGamma = -1;
	double p = 0;

	int nRoot = 0;


	int begining = indiceBusBegin[bus];
	int nC = nChild[bus];

	bool goodSol = false;
	k2 = sqrt(2.0 / (nC + 1));
	if (index == 0)
	{
		/*if (bus == 0) {
			goodSol = true;
			x1 = 0;
			x2 = 0;
			x4 = 0;
			x3 = 1 / k2;
			gamma = 0;
		}
		else {*/

		c1 = -2 * Chat[beginChat];
		c2 = -2 * Chat[beginChat + 1];
		c4 = -2 * Chat[beginChat + 2];
		c3 = -2 * Chat[beginChat + 3] / k2;

		c1122 = c1 * c1 + c2 * c2;
		x3min = Vbound[bus];
		x3max = Vbound[bus + nBus];

		// case without constraint

		x1 = -c1 / 2;
		x2 = -c2 / 2;
		x3 = -c3 / 2;
		x4 = -c4 / 2;

		lambdaUp = 0;
		lambdaLo = 0;
		if (x3 < x3min) {
			x3 = x3min;
			lambdaLo = (2 * x3 + c3);
		}
		else if (x3 > x3max) {
			x3 = x3max;
			lambdaUp = -(2 * x3 + c3);
		}
		gamma = k2 * x4 - (x1 * x1 + x2 * x2) / x3; // ce n'est pas vraiment gamma, doit être positif

		if (gamma >= 0) {
			// the solution is good !
			goodSol = true;
		}
		else {
			if (c1122 == 0) {
				x4 = 0;
				goodSol = true;
			}
			if (gamma > bestGamma) {
				typeSol = 1;
				bestGamma = gamma;
			}
		}
		//}
		if (!goodSol) {
			x3 = x3max;

			coefPoly2[0] = 2 * (c4 / (k2 * x3) + 1);
			coefPoly2[1] = 1 / x3;
			coefPoly2[0] = coefPoly2[0] * k2 * k2 / (4 * c1122);
			coefPoly2[1] = coefPoly2[1] * k2 * k2 / (4 * c1122);
			nRoot = resolveRealPolynome3without2termGPU(root2, coefPoly2[0], coefPoly2[1]);

			for (int n = 0; n < nRoot; n++) {
				p = root2[n];

				x1 = p * c1 * x3;
				x2 = p * c2 * x3;
				x4 = (x1 * x1 + x2 * x2) / (x3 * k2);
				gamma = (2 * x4 + c4) / k2;
				lambdaUp = -(2 * x3 + c3 - gamma * (x1 * x1 + x2 * x2) / (x3 * x3));

				if (gamma >= 0 && lambdaUp >= 0) {
					goodSol = true;
					break;
				}
				if (gamma > bestGamma && lambdaUp > bestGamma) {
					typeSol = 2;
					bestGamma = min(gamma, lambdaUp);
					BestRoot = n;
				}

			}
		}
			// case x3 = x3min lambdaUp = 0
		if (!goodSol) {
			x3 = x3min;

			coefPoly2[0] = 2 * (c4 / (k2 * x3) + 1);
			coefPoly2[1] = 1 / x3;
			coefPoly2[0] = coefPoly2[0] * k2 * k2 / (4 * c1122);
			coefPoly2[1] = coefPoly2[1] * k2 * k2 / (4 * c1122);
			nRoot = resolveRealPolynome3without2termGPU(root3, coefPoly2[0], coefPoly2[1]);

			for (int n = 0; n < nRoot; n++) {
				p = root3[n];
				x1 = p * c1 * x3;
				x2 = p * c2 * x3;
				x4 = (x1 * x1 + x2 * x2) / (x3 * k2);
				gamma = (2 * x4 + c4) / k2;
				lambdaLo = (2 * x3 + c3 - gamma * (x1 * x1 + x2 * x2) / (x3 * x3));

				if (gamma >= 0 && lambdaLo >= 0) {
					// the solution is good !
					goodSol = true;
					break;
				}
				if (gamma > bestGamma && lambdaLo > bestGamma) {
					typeSol = 3;
					bestGamma = min(gamma, lambdaLo);
					BestRoot = n;
				}
			}
		}
			// case xmin<x3<xmax lambdaLo = 0 lambdaUp = 0
		if (!goodSol) {

			coefPoly3[0] = c1122 / k2 * (2 * c3 / k2 - c4);
			coefPoly3[1] = (c3 - 2 * c4 / k2);
			coefPoly3[2] = -1;
			coefPoly3[0] = coefPoly3[0] * k2 * k2 / (c1122 * c1122);
			coefPoly3[1] = coefPoly3[1] * k2 * k2 / (c1122 * c1122);
			coefPoly3[2] = coefPoly3[2] * k2 * k2 / (c1122 * c1122);

			nRoot = resvolveRealPolynome4without2termGPU(root4, coefPoly3[0], coefPoly3[1], coefPoly3[2], Lagrange);

			for (int n = 0; n < nRoot; n++) {
				p = root4[n];
				x3 = -(c1122 * p + 2 * c3) / (2 * (c1122 * p * p + 2));
				x1 = p * c1 * x3;
				x2 = p * c2 * x3;
				x4 = (x1 * x1 + x2 * x2) / (x3 * k2);
				gamma = (2 * x4 + c4) / k2;

				if (gamma >= 0 && x3 <= x3max && x3 >= x3min) {
					// the solution is good !
					goodSol = true;
					break;
				}
				if (gamma > bestGamma && (x3max - x3) > bestGamma && (x3 - x3min) > bestGamma) {
					typeSol = 4;
					bestGamma = min(min(gamma, (x3max - x3)), (x3 - x3min));
					BestRoot = n;
				}
			}
		}
		if (!goodSol) {

			if (typeSol == 1) {
				// case without constraint
				x1 = -c1 / 2;
				x2 = -c2 / 2;
				x3 = -c3 / 2;
				x3 = (x3max - x3) * (x3 > x3max) + (x3min - x3) * (x3min > x3) + x3;
				x4 = -c4 / 2; // ou  (x1 * x1 + x2 * x2) / (k2 * x3)
			}
			else {
				if (typeSol == 2) {
					x3 = x3max;
					p = root2[BestRoot];
				}
				else if (typeSol == 3) {
					x3 = x3min;
					p = root3[BestRoot];
				}
				else if (typeSol == 4) {
					p = root4[BestRoot];
					x3 = -(c1122 * p + 2 * c3) / (2 * (c1122 * p * p + 2));
					x3 = (x3max - x3) * (x3 > x3max) + (x3min - x3) * (x3min > x3) + x3;
				}
				x1 = p * c1 * x3;
				x2 = p * c2 * x3;
				x4 = (x1 * x1 + x2 * x2) / (x3 * k2);
			}
		}

		X[begining] = x1;
		X[begining + 1] = x2;
		X[begining + 2] = x4;
		X[begining + 3] = x3 * k2;

	}
	int nb = nAgentByBus[bus];
	int beginAgent = CoresAgentBusBegin[bus];
	for (int i = index; i < nb; i += step) {
		int n = CoresAgentBus[i + beginAgent];
		float ub = Pmax[n];
		float lb = Pmin[n];
		float pn = (rho * Chat[beginChat + 4 + i] - Cost2[n]) / (Cost1[n] + rho);
		pn = (ub - pn) * (pn > ub) + (lb - pn) * (pn < lb) + pn;


		ub = Pmax[n + nAgent];
		lb = Pmin[n + nAgent];
		float qn = (rho * Chat[beginChat + 4 + nb + i] - Cost2[n + nAgent]) / (Cost1[n + nAgent] + rho);
		qn = (ub - qn) * (qn > ub) + (lb - qn) * (qn < lb) + qn;

		// pn & qn
		X[begining + 4 + i] = pn;
		X[begining + 4 + nb + i] = qn;
	}


	// X =  {Pi, Qi, vi, li, pi, qi, vAi, (Pci, Qci, lci) for all child Ci}	

}

__global__ void updateXOPFADMM(float* X, float* Chat, float* Vbound, float* PnMoy, float* nAgentByBus, float* nChild, float* indiceBusBegin, int nBus, bool Lagrange) {

	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int step = blockDim.x * gridDim.x;
	double x1, x2, x3, x4, c1, c2, c3, c4, lambdaLo, lambdaUp, x3min, x3max, gamma, k2; // double peut �tre necessaire
	double c1122; // c3 : voltage -> indice + 3, c4 : current -> indice + 2;
	double coefPoly2[2];
	double root2[4];
	double root3[4];
	double root4[4];
	double coefPoly3[3];
	int typeSol = 0;
	int BestRoot = 0;
	double bestGamma = -1;
	double p = 0;

	int nRoot = 0;

	for (int b = index + 1; b < nBus; b += step) {
		int begining = indiceBusBegin[b];
		int nC = nChild[b];

		bool goodSol = false;
		k2 = sqrt(2.0 / (nC + 1));
		/*if (b == 0) { // slack bus divergent, mais on n'y peut rien
			goodSol = true;
			x1 = 0;
			x2 = 0;
			x4 = 0;
			x3 = 1 / k2;
			gamma = 0;
		}*/
		//else {

		c1 = -2 * Chat[b];
		c2 = -2 * Chat[b + nBus];
		c4 = -2 * Chat[b + 2 * nBus];
		c3 = -2 * Chat[b + 3 * nBus] / k2;

		c1122 = c1 * c1 + c2 * c2;
		x3min = Vbound[b];
		x3max = Vbound[b + nBus];

		// case without constraint

		x1 = -c1 / 2;
		x2 = -c2 / 2;
		x3 = -c3 / 2;
		x4 = -c4 / 2;

		lambdaUp = 0;
		lambdaLo = 0;
		if (x3 < x3min) {
			x3 = x3min;
			lambdaLo = (2 * x3 + c3);
		}
		else if (x3 > x3max) {
			x3 = x3max;
			lambdaUp = -(2 * x3 + c3);
		}
		gamma = k2 * x4 - (x1 * x1 + x2 * x2) / x3; // ce n'est pas vraiment gamma, doit être positif

		if (gamma >= 0) {
			// the solution is good !
			goodSol = true;
		}
		else {
			if (c1122 == 0) {
				x4 = 0;
				goodSol = true;
			}
			if (gamma > bestGamma) {
				typeSol = 1;
				bestGamma = gamma;
			}
		}
		//}
		if (!goodSol) {
			x3 = x3max;

			coefPoly2[0] = 2 * (c4 / (k2 * x3) + 1);
			coefPoly2[1] = 1 / x3;
			coefPoly2[0] = coefPoly2[0] * k2 * k2 / (4 * c1122);
			coefPoly2[1] = coefPoly2[1] * k2 * k2 / (4 * c1122);
			nRoot = resolveRealPolynome3without2termGPU(root2, coefPoly2[0], coefPoly2[1]);

			for (int n = 0; n < nRoot; n++) {
				p = root2[n];

				x1 = p * c1 * x3;
				x2 = p * c2 * x3;
				x4 = (x1 * x1 + x2 * x2) / (x3 * k2);
				gamma = (2 * x4 + c4) / k2;
				lambdaUp = -(2 * x3 + c3 - gamma * (x1 * x1 + x2 * x2) / (x3 * x3));

				if (gamma >= 0 && lambdaUp >= 0) {
					goodSol = true;
					break;
				}
				if (gamma > bestGamma && lambdaUp > bestGamma) {
					typeSol = 2;
					bestGamma = min(gamma, lambdaUp);
					BestRoot = n;
				}

			}
		}
		// case x3 = x3min lambdaUp = 0
		if (!goodSol) {
			x3 = x3min;

			coefPoly2[0] = 2 * (c4 / (k2 * x3) + 1);
			coefPoly2[1] = 1 / x3;
			coefPoly2[0] = coefPoly2[0] * k2 * k2 / (4 * c1122);
			coefPoly2[1] = coefPoly2[1] * k2 * k2 / (4 * c1122);
			nRoot = resolveRealPolynome3without2termGPU(root2, coefPoly2[0], coefPoly2[1]);

			for (int n = 0; n < nRoot; n++) {
				p = root3[n];
				x1 = p * c1 * x3;
				x2 = p * c2 * x3;
				x4 = (x1 * x1 + x2 * x2) / (x3 * k2);
				gamma = (2 * x4 + c4) / k2;
				lambdaLo = (2 * x3 + c3 - gamma * (x1 * x1 + x2 * x2) / (x3 * x3));

				if (gamma >= 0 && lambdaLo >= 0) {
					// the solution is good !
					goodSol = true;
					break;
				}
				if (gamma > bestGamma && lambdaLo > bestGamma) {
					typeSol = 3;
					bestGamma = min(gamma, lambdaLo);
					BestRoot = n;
				}
			}
		}
		// case xmin<x3<xmax lambdaLo = 0 lambdaUp = 0
		if (!goodSol) {

			coefPoly3[0] = c1122 / k2 * (2 * c3 / k2 - c4);
			coefPoly3[1] = (c3 - 2 * c4 / k2);
			coefPoly3[2] = -1;
			coefPoly3[0] = coefPoly3[0] * k2 * k2 / (c1122 * c1122);
			coefPoly3[1] = coefPoly3[1] * k2 * k2 / (c1122 * c1122);
			coefPoly3[2] = coefPoly3[2] * k2 * k2 / (c1122 * c1122);

			nRoot = resvolveRealPolynome4without2termGPU(root4, coefPoly3[0], coefPoly3[1], coefPoly3[2], Lagrange);

			for (int n = 0; n < nRoot; n++) {
				p = root4[n];
				x3 = -(c1122 * p + 2 * c3) / (2 * (c1122 * p * p + 2));
				x1 = p * c1 * x3;
				x2 = p * c2 * x3;
				x4 = (x1 * x1 + x2 * x2) / (x3 * k2);
				gamma = (2 * x4 + c4) / k2;

				if (gamma >= 0 && x3 <= x3max && x3 >= x3min) {
					// the solution is good !
					goodSol = true;
					break;
				}
				if (gamma > bestGamma && (x3max - x3) > bestGamma && (x3 - x3min) > bestGamma) {
					typeSol = 4;
					bestGamma = min(min(gamma, (x3max - x3)), (x3 - x3min));
					BestRoot = n;
				}
			}
		}


		if (!goodSol) {

			if (typeSol == 1) {
				// case without constraint
				x1 = -c1 / 2;
				x2 = -c2 / 2;
				x3 = -c3 / 2;
				x3 = (x3max - x3) * (x3 > x3max) + (x3min - x3) * (x3min > x3) + x3;
				x4 = -c4 / 2; // ou  (x1 * x1 + x2 * x2) / (k2 * x3)
			}
			else {
				if (typeSol == 2) {
					x3 = x3max;
					p = root2[BestRoot];
				}
				else if (typeSol == 3) {
					x3 = x3min;
					p = root3[BestRoot];
				}
				else if (typeSol == 4) {
					p = root4[BestRoot];
					x3 = -(c1122 * p + 2 * c3) / (2 * (c1122 * p * p + 2));
					x3 = (x3max - x3) * (x3 > x3max) + (x3min - x3) * (x3min > x3) + x3;
				}
				x1 = p * c1 * x3;
				x2 = p * c2 * x3;
				x4 = (x1 * x1 + x2 * x2) / (x3 * k2);
			}
		}
		// X =  {Pi, Qi, vi, li, pi, qi, vAi, (Pci, Qci, lci) for all child Ci}	
		X[begining] = x1;
		X[begining + 1] = x2;
		X[begining + 2] = x4;
		X[begining + 3] = x3 * k2;

	}

	for (int b = index; b < nBus; b += step) {
		int begining = indiceBusBegin[b];
		X[begining + 4] = PnMoy[b] * nAgentByBus[b];
		X[begining + 5] = PnMoy[b + nBus] * nAgentByBus[b];
	}
}


__global__ void updateXEndoMarket(float* X, float* Chat, float* Vbound, float* nChild, float* CoresChatBegin, float* indiceBusBegin, int nBus) {
	
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int step = blockDim.x * gridDim.x;
	
	double x1, x2, x3, x4, c1, c2, c3, c4, lambdaLo, lambdaUp, x3min, x3max, gamma, k2; // double peut �tre necessaire
	double c1122; // c3 : voltage -> indice + 3, c4 : current -> indice + 2;
	double coefPoly2[2];
	double root2[4];
	double root3[4];
	double root4[4];
	double coefPoly3[3];
	int typeSol = 0;
	int BestRoot = 0;
	double bestGamma = -1;
	double p = 0;

	int nRoot = 0;


	bool goodSol = false;
	
	for (int bus = index + 1; bus < nBus; bus += step) {
		int beginChat = CoresChatBegin[bus];
		int begining = indiceBusBegin[bus];
		int nC = nChild[bus];
		k2 = sqrt(2.0 / (nC + 1));

		c1 = -2 * Chat[beginChat];
		c2 = -2 * Chat[beginChat + 1];
		c4 = -2 * Chat[beginChat + 2];
		c3 = -2 * Chat[beginChat + 3] / k2;

		c1122 = c1 * c1 + c2 * c2;
		x3min = Vbound[bus];
		x3max = Vbound[bus + nBus];

		// case without constraint

		x1 = -c1 / 2;
		x2 = -c2 / 2;
		x3 = -c3 / 2;
		x4 = -c4 / 2;

		lambdaUp = 0;
		lambdaLo = 0;
		if (x3 < x3min) {
			x3 = x3min;
			lambdaLo = (2 * x3 + c3);
		}
		else if (x3 > x3max) {
			x3 = x3max;
			lambdaUp = -(2 * x3 + c3);
		}

		gamma = k2 * x4 - (x1 * x1 + x2 * x2) / x3; // ce n'est pas vraiment gamma, doit être positif

		if (gamma >= 0) {
			// the solution is good !
			goodSol = true;
		}
		else {
			if (c1122 == 0) {
				x4 = 0;
				goodSol = true;
			}
			else if (gamma > bestGamma) {
				typeSol = 1;
				bestGamma = gamma;
			}
		}

		if (!goodSol) {
			x3 = x3max;

			coefPoly2[0] = 2 * (c4 / (k2 * x3) + 1);
			coefPoly2[1] = 1 / x3;
			coefPoly2[0] = coefPoly2[0] * k2 * k2 / (4 * c1122);
			coefPoly2[1] = coefPoly2[1] * k2 * k2 / (4 * c1122);
			nRoot = resolveRealPolynome3without2termGPU(root2, coefPoly2[0], coefPoly2[1]);

			for (int n = 0; n < nRoot; n++) {
				p = root2[n];

				x1 = p * c1 * x3;
				x2 = p * c2 * x3;
				x4 = (x1 * x1 + x2 * x2) / (x3 * k2);
				gamma = (2 * x4 + c4) / k2;
				lambdaUp = -(2 * x3 + c3 - gamma * (x1 * x1 + x2 * x2) / (x3 * x3));

				if (gamma >= 0 && lambdaUp >= 0) {
					goodSol = true;
					break;
				}
				if (gamma > bestGamma && lambdaUp > bestGamma) {
					typeSol = 2;
					bestGamma = min(gamma, lambdaLo);
					BestRoot = n;
				}

			}
			// case x3 = x3min lambdaUp = 0
		}
		if (!goodSol) {
			x3 = x3min;

			coefPoly2[0] = 2 * (c4 / (k2 * x3) + 1);
			coefPoly2[1] = 1 / x3;
			coefPoly2[0] = coefPoly2[0] * k2 * k2 / (4 * c1122);
			coefPoly2[1] = coefPoly2[1] * k2 * k2 / (4 * c1122);
			nRoot = resolveRealPolynome3without2termGPU(root3, coefPoly2[0], coefPoly2[1]);

			for (int n = 0; n < nRoot; n++) {
				p = root3[n];
				x1 = p * c1 * x3;
				x2 = p * c2 * x3;
				x4 = (x1 * x1 + x2 * x2) / (x3 * k2);
				gamma = (2 * x4 + c4) / k2;
				lambdaLo = (2 * x3 + c3 - gamma * (x1 * x1 + x2 * x2) / (x3 * x3));

				if (gamma >= 0 && lambdaLo >= 0) {
					// the solution is good !
					goodSol = true;
					break;
				}
				if (gamma > bestGamma && lambdaLo > bestGamma) {
					typeSol = 3;
					bestGamma = min(gamma, lambdaLo);
					BestRoot = n;
				}
			}
		}
		// case xmin<x3<xmax lambdaLo = 0 lambdaUp = 0
		if (!goodSol) {
			coefPoly3[0] = c1122 / k2 * (2 * c3 / k2 - c4);
			coefPoly3[1] = (c3 - 2 * c4 / k2);
			coefPoly3[2] = -1;
			coefPoly3[0] = coefPoly3[0] * k2 * k2 / (c1122 * c1122);
			coefPoly3[1] = coefPoly3[1] * k2 * k2 / (c1122 * c1122);
			coefPoly3[2] = coefPoly3[2] * k2 * k2 / (c1122 * c1122);

			nRoot = resvolveRealPolynome4without2termGPU(root4, coefPoly3[0], coefPoly3[1], coefPoly3[2]);

			for (int n = 0; n < nRoot; n++) {
				p = root4[n];
				x3 = -(c1122 * p + 2 * c3) / (2 * (c1122 * p * p + 2));
				x1 = p * c1 * x3;
				x2 = p * c2 * x3;
				x4 = (x1 * x1 + x2 * x2) / (x3 * k2);
				gamma = (2 * x4 + c4) / k2;

				if (gamma >= 0 && x3 <= x3max && x3 >= x3min) {
					// the solution is good !
					goodSol = true;
					break;
				}
				if (gamma > bestGamma && (x3max - x3) > bestGamma && (x3 - x3min) > bestGamma) {
					typeSol = 4;
					bestGamma = min(min(gamma, (x3max - x3)), (x3 - x3min));
					BestRoot = n;
				}
			}
		}

		if (!goodSol) {

			if (typeSol == 1) {
				// case without constraint
				x1 = -c1 / 2;
				x2 = -c2 / 2;
				x3 = -c3 / 2;
				x3 = (x3max - x3) * (x3 > x3max) + (x3min - x3) * (x3min > x3) + x3;
				x4 = -c4 / 2; // ou  (x1 * x1 + x2 * x2) / (k2 * x3)
			}
			else {
				if (typeSol == 2) {
					x3 = x3max;
					p = root2[BestRoot];
				}
				else if (typeSol == 3) {
					x3 = x3min;
					p = root3[BestRoot];
				}
				else if (typeSol == 4) {
					p = root4[BestRoot];
					x3 = -(c1122 * p + 2 * c3) / (2 * (c1122 * p * p + 2));
					x3 = (x3max - x3) * (x3 > x3max) + (x3min - x3) * (x3min > x3) + x3;
				}
				x1 = p * c1 * x3;
				x2 = p * c2 * x3;
				x4 = (x1 * x1 + x2 * x2) / (x3 * k2);
			}
		}

		X[begining] = x1;
		X[begining + 1] = x2;
		X[begining + 2] = x4;
		X[begining + 3] = x3 * k2;
		


	}
	

}

__global__ void updateXOPFADMMCons(float* X, float* Pn, float* Chat, float* Vbound, float* nAgentByBus, float* nChild, float* indiceBusBegin, float* CoresChatBegin,
	float* CoresAgentBusBegin, float* CoresAgentBus, float* Cost1, float* Cost2, float* Pmin, float* Pmax, float rho, int losstype, int nBus, int nAgent, bool Lagrange) {

	int bus = blockIdx.x + 1; // pas faire le bus 0
	int index = threadIdx.x;
	int step = blockDim.x;
	
	double x1, x2, x3, x4, c1, c2, c3, c4, lambdaLo, lambdaUp, x3min, x3max, gamma, k2; // double peut �tre necessaire
	double c1122; // c3 : voltage -> indice + 3, c4 : current -> indice + 2;
	double coefPoly2[2];
	double root2[4];
	double root3[4];
	double root4[4];
	double coefPoly3[3];
	int typeSol = 0;
	int BestRoot = 0;
	double bestGamma = -1;
	double p = 0;

	int nRoot = 0;


	int begining = indiceBusBegin[bus];
	int nC = nChild[bus];
	int beginChat = CoresChatBegin[bus];

	bool goodSol = false;
	k2 = sqrt(2.0 / (nC + 1));
	if (bus < nBus) {
		if (index == 0)
		{
			c1 = -2 * Chat[beginChat];
			c2 = -2 * Chat[beginChat + 1];
			c4 = -2 * Chat[beginChat + 2];
			c3 = -2 * Chat[beginChat + 3] / k2;

			c1122 = c1 * c1 + c2 * c2;
			x3min = Vbound[bus];
			x3max = Vbound[bus + nBus];

			// case without constraint

			x1 = -c1 / 2;
			x2 = -c2 / 2;
			x3 = -c3 / 2;
			x4 = -c4 / 2;

			lambdaUp = 0;
			lambdaLo = 0;
			if (x3 < x3min) {
				x3 = x3min;
				lambdaLo = (2 * x3 + c3);
			}
			else if (x3 > x3max) {
				x3 = x3max;
				lambdaUp = -(2 * x3 + c3);
			}
			gamma = k2 * x4 - (x1 * x1 + x2 * x2) / x3; // ce n'est pas vraiment gamma, doit être positif

			if (gamma >= 0) {
				// the solution is good !
				goodSol = true;
			}
			else {
				if (c1122 == 0) {
					x4 = 0;
					goodSol = true;
				}
				if (gamma > bestGamma) {
					typeSol = 1;
					bestGamma = gamma;
				}
			}
			//}
			if (!goodSol) {
				x3 = x3max;

				coefPoly2[0] = 2 * (c4 / (k2 * x3) + 1);
				coefPoly2[1] = 1 / x3;
				coefPoly2[0] = coefPoly2[0] * k2 * k2 / (4 * c1122);
				coefPoly2[1] = coefPoly2[1] * k2 * k2 / (4 * c1122);
				nRoot = resolveRealPolynome3without2termGPU(root2, coefPoly2[0], coefPoly2[1]);

				for (int n = 0; n < nRoot; n++) {
					p = root2[n];

					x1 = p * c1 * x3;
					x2 = p * c2 * x3;
					x4 = (x1 * x1 + x2 * x2) / (x3 * k2);
					gamma = (2 * x4 + c4) / k2;
					lambdaUp = -(2 * x3 + c3 - gamma * (x1 * x1 + x2 * x2) / (x3 * x3));

					if (gamma >= 0 && lambdaUp >= 0) {
						goodSol = true;
						break;
					}
					if (gamma > bestGamma && lambdaUp > bestGamma) {
						typeSol = 2;
						bestGamma = min(gamma, lambdaUp);
						BestRoot = n;
					}

				}
			}
			// case x3 = x3min lambdaUp = 0
			if (!goodSol) {
				x3 = x3min;

				coefPoly2[0] = 2 * (c4 / (k2 * x3) + 1);
				coefPoly2[1] = 1 / x3;
				coefPoly2[0] = coefPoly2[0] * k2 * k2 / (4 * c1122);
				coefPoly2[1] = coefPoly2[1] * k2 * k2 / (4 * c1122);
				nRoot = resolveRealPolynome3without2termGPU(root3, coefPoly2[0], coefPoly2[1]);

				for (int n = 0; n < nRoot; n++) {
					p = root3[n];
					x1 = p * c1 * x3;
					x2 = p * c2 * x3;
					x4 = (x1 * x1 + x2 * x2) / (x3 * k2);
					gamma = (2 * x4 + c4) / k2;
					lambdaLo = (2 * x3 + c3 - gamma * (x1 * x1 + x2 * x2) / (x3 * x3));

					if (gamma >= 0 && lambdaLo >= 0) {
						// the solution is good !
						goodSol = true;
						break;
					}
					if (gamma > bestGamma && lambdaLo > bestGamma) {
						typeSol = 3;
						bestGamma = min(gamma, lambdaLo);
						BestRoot = n;
					}
				}
			}
			// case xmin<x3<xmax lambdaLo = 0 lambdaUp = 0
			if (!goodSol) {

				coefPoly3[0] = c1122 / k2 * (2 * c3 / k2 - c4);
				coefPoly3[1] = (c3 - 2 * c4 / k2);
				coefPoly3[2] = -1;
				coefPoly3[0] = coefPoly3[0] * k2 * k2 / (c1122 * c1122);
				coefPoly3[1] = coefPoly3[1] * k2 * k2 / (c1122 * c1122);
				coefPoly3[2] = coefPoly3[2] * k2 * k2 / (c1122 * c1122);

				nRoot = resvolveRealPolynome4without2termGPU(root4, coefPoly3[0], coefPoly3[1], coefPoly3[2], Lagrange);

				for (int n = 0; n < nRoot; n++) {
					p = root4[n];
					x3 = -(c1122 * p + 2 * c3) / (2 * (c1122 * p * p + 2));
					x1 = p * c1 * x3;
					x2 = p * c2 * x3;
					x4 = (x1 * x1 + x2 * x2) / (x3 * k2);
					gamma = (2 * x4 + c4) / k2;

					if (gamma >= 0 && x3 <= x3max && x3 >= x3min) {
						// the solution is good !
						goodSol = true;
						break;
					}
					if (gamma > bestGamma && (x3max - x3) > bestGamma && (x3 - x3min) > bestGamma) {
						typeSol = 4;
						bestGamma = min(min(gamma, (x3max - x3)), (x3 - x3min));
						BestRoot = n;
					}
				}
			}
			if (!goodSol) {

				if (typeSol == 1) {
					// case without constraint
					x1 = -c1 / 2;
					x2 = -c2 / 2;
					x3 = -c3 / 2;
					x3 = (x3max - x3) * (x3 > x3max) + (x3min - x3) * (x3min > x3) + x3;
					x4 = -c4 / 2; // ou  (x1 * x1 + x2 * x2) / (k2 * x3)
				}
				else {
					if (typeSol == 2) {
						x3 = x3max;
						p = root2[BestRoot];
					}
					else if (typeSol == 3) {
						x3 = x3min;
						p = root3[BestRoot];
					}
					else if (typeSol == 4) {
						p = root4[BestRoot];
						x3 = -(c1122 * p + 2 * c3) / (2 * (c1122 * p * p + 2));
						x3 = (x3max - x3) * (x3 > x3max) + (x3min - x3) * (x3min > x3) + x3;
					}
					x1 = p * c1 * x3;
					x2 = p * c2 * x3;
					x4 = (x1 * x1 + x2 * x2) / (x3 * k2);
				}
			}

			X[begining] = x1;
			X[begining + 1] = x2;
			X[begining + 2] = x4;
			X[begining + 3] = x3 * k2;

		}
		int nb = nAgentByBus[bus];
		int beginAgent = CoresAgentBusBegin[bus];
		for (int i = index; i < nb; i += step) {
			int n = CoresAgentBus[i + beginAgent];
			float ub = Pmax[n];
			float lb = Pmin[n];
			float pn = (rho * Chat[beginChat + 4 + i] - Cost2[n]) / (Cost1[n] + rho);
			pn = (ub - pn) * (pn > ub) + (lb - pn) * (pn < lb) + pn;


			ub = Pmax[n + nAgent];
			lb = Pmin[n + nAgent];
			float qn = (rho * Chat[beginChat + 4 + nb + i] - Cost2[n + nAgent]) / (Cost1[n + nAgent] + rho);
			qn = (ub - qn) * (qn > ub) + (lb - qn) * (qn < lb) + qn;

			// pn & qn
			X[begining + 4 + i] = pn;
			X[begining + 4 + nb + i] = qn;
			Pn[n] = pn;
			Pn[n + nAgent] = qn;
		}
	}
	else { 
		if (index == 0) { 
			// bus des pertes
			float pn = ( rho * Chat[beginChat] - Cost2[0]) / (Cost1[0] + rho);
			float qn = ( rho * Chat[beginChat + 1] - Cost2[nAgent]) / (Cost1[nAgent] + rho);
			int offset = (losstype == 0) * nAgent + (losstype == 1) * 1;
			X[begining] = pn;
			X[begining + offset] = qn;
		}
		// puissance sur le bus 0
		int nb = nAgentByBus[0];
		int beginAgent = CoresAgentBusBegin[0];
		int begin0 = indiceBusBegin[0];
		int beginChat = CoresChatBegin[0];

		for (int i = index; i < nb; i += step) {
			int n = CoresAgentBus[i + beginAgent];
			float ub = Pmax[n];
			float lb = Pmin[n];
			float pn = (rho * Chat[beginChat + 4 + i] - Cost2[n]) / (Cost1[n] + rho);
			pn = (ub - pn) * (pn > ub) + (lb - pn) * (pn < lb) + pn;


			ub = Pmax[n + nAgent];
			lb = Pmin[n + nAgent];
			float qn = (rho * Chat[beginChat + 4 + nb + i] - Cost2[n + nAgent]) / (Cost1[n + nAgent] + rho);
			qn = (ub - qn) * (qn > ub) + (lb - qn) * (qn < lb) + qn;

			// pn & qn
			X[begin0 + 4 + i] = pn;
			X[begin0 + 4 + nb + i] = qn;
			Pn[n] = pn;
			Pn[n + nAgent] = qn;
		}



	}
	


	// X =  {Pi, Qi, vi, li, pi, qi, vAi, (Pci, Qci, lci) for all child Ci}	

}


__global__ void updateXPnOPFADMMCons(float* X, float* Pn, float* Chat, float* nAgentByBus, float* indiceBusBegin, float* CoresChatBegin,
	float* CoresAgentBusBegin, float* CoresAgentBus, float* Cost1, float* Cost2, float* Pmin, float* Pmax, float rho, int losstype, int nBus, int nAgent) {

	int bus = blockIdx.x;
	int index = threadIdx.x;
	int step = blockDim.x;

	int begining = indiceBusBegin[bus];
	int beginChat = CoresChatBegin[bus];
		
	if (bus < nBus) {
		int nb = nAgentByBus[bus];
		int beginAgent = CoresAgentBusBegin[bus];
		for (int i = index; i < nb; i += step) {
			int n = CoresAgentBus[i + beginAgent];
			float ub = Pmax[n];
			float lb = Pmin[n];
			float pn = (rho * Chat[beginChat + 4 + i] - Cost2[n]) / (Cost1[n] + rho);
			pn = (ub - pn) * (pn > ub) + (lb - pn) * (pn < lb) + pn;


			ub = Pmax[n + nAgent];
			lb = Pmin[n + nAgent];
			float qn = (rho * Chat[beginChat + 4 + nb + i] - Cost2[n + nAgent]) / (Cost1[n + nAgent] + rho);
			qn = (ub - qn) * (qn > ub) + (lb - qn) * (qn < lb) + qn;

			// pn & qn
			X[begining + 4 + i] = pn;
			X[begining + 4 + nb + i] = qn;
			Pn[n] = pn;
			Pn[n + nAgent] = qn;
		}
	}
	else {
		if (index == 0) {
			// bus des pertes
			float pn = (rho * Chat[beginChat] - Cost2[0]) / (Cost1[0] + rho);
			float qn = (rho * Chat[beginChat + 1] - Cost2[nAgent]) / (Cost1[nAgent] + rho);
			int offset = (losstype == 0) * nAgent + (losstype == 1) * 1;
			X[begining] = pn;
			X[begining + offset] = qn;
		}
	
	}

	// X =  {Pi, Qi, vi, li, pi, qi, vAi, (Pci, Qci, lci) for all child Ci}	

}



__global__ void updateXPn(float* X, float* Pn, float* P, float* nVoisin, float* indiceBusBegin, float* nAgentByBus, float* CoresAgentBusBegin, float* CoresAgentBus, int lossType, int nAgent, int nBus) {
	
	int bus = blockIdx.x; // un bloc par bus
	int index = threadIdx.x;
	int begin = indiceBusBegin[bus];
	int step = blockDim.x;
	
	if (bus == nBus) { // bus des pertes
		if (index == 0) {
			int indice = (lossType == 0) * (nAgent - 1) + 1;
			float pn = P[0] * nVoisin[0];
			float qn = P[nAgent] * nVoisin[nAgent];
			Pn[0] = pn;
			Pn[nAgent] = qn;
			X[begin] = pn;
			X[begin + indice] = qn;
		}
	}
	else {
		
		int Nb = nAgentByBus[bus];
		int beginAgent = CoresAgentBusBegin[bus];
		for (int In = index; In < Nb; In += step) {
			int n = CoresAgentBus[In + beginAgent];

			float pn = P[n] * nVoisin[n];
			float qn = P[n + nAgent] * nVoisin[n + nAgent];

			Pn[n] = pn;
			Pn[nAgent + n] = qn;
			// pn & qn
			X[begin + 4 + In] = pn;
			X[begin + 4 + Nb + In] = qn;
		}

	}



}

__global__ void communicateX(float* X, float* nChild, float* Ancestor, float* Childs, float* indiceBusBegin, float* indiceChildBegin, float* nAgentByBus, int nBus) {
	int bus = blockIdx.x;
	int index = threadIdx.x;
	int step = blockDim.x;

	int indice = indiceBusBegin[bus];
	int indiceChild = (bus < (nBus - 1)) ? indiceChildBegin[bus] : 0;
	int nb = nChild[bus];
	int nAgent = nAgentByBus[bus];

	if (index == 0) { // Vai
		int Ai = Ancestor[bus];
		int indiceAi = bus > 0 ? indiceBusBegin[Ai] : 0;
		X[indice + 4 + 2 * nAgent] = bus > 0 ? X[indiceAi + 3] : 1;
	}


	for (int voisin = index; voisin < nb; voisin += step) { //  coalescent en ecriture mais pas en lecture 
		int c = Childs[indiceChild + voisin];
		int indiceBusChild = indiceBusBegin[c];
		X[indice + 5 + 2 * nAgent + voisin] = X[indiceBusChild];
		X[indice + 5 + 2 * nAgent + nb + voisin] = X[indiceBusChild + 1];
		X[indice + 5 + 2 * nAgent + 2 * nb + voisin] = X[indiceBusChild + 2];
	}

}
__global__ void communicateX(float* X, float* nChild, float* Ancestor, float* Childs, float* indiceBusBegin, float* indiceChildBegin, int nBus) {
	int bus = blockIdx.x;
	int index = threadIdx.x;
	int step = blockDim.x;

	int indice = indiceBusBegin[bus];
	int indiceChild = (bus < (nBus - 1)) ? indiceChildBegin[bus] : 0;
	int nb = nChild[bus];

	if (index == 0) { // Vai
		int Ai = Ancestor[bus];
		int indiceAi = bus > 0 ? indiceBusBegin[Ai] : 0;
		X[indice + 6] = bus > 0 ? X[indiceAi + 3] : 1;
	}


	for (int voisin = index; voisin < nb; voisin += step) { // pas coalescent et am�liorable
		int c = Childs[indiceChild + voisin];
		int indiceBusChild = indiceBusBegin[c];
		X[indice + 7 + voisin] = X[indiceBusChild]; // un truc du style X[indice + 7 + voisin], X[indice + 7 + nb + voisin] et X[indice + 7 + 2*nb + voisin] serait coalescent
		X[indice + 7 + nb + voisin] = X[indiceBusChild + 1];
		X[indice + 7 + 2 * nb + voisin] = X[indiceBusChild + 2];
	}
	/*
	for (int i = 0; i < _nBus; i++) {

			if (i > 0) {
				int Ai = Ancestor.get(i, 0);
				X[i].set(6, 0, X[Ai].get(2, 0));
			}

			int m = nChild.get(i, 0);
			for (int j = 0; j < m; j++) {
				int c = Childs[i].get(j, 0);
				X[i].set(7 + 3 * j, 0, X[c].get(0, 0));
				X[i].set(8 + 3 * j, 0, X[c].get(1, 0));
				X[i].set(9 + 3 * j, 0, X[c].get(3, 0));

			}
		}

	*/


}

__global__ void communicateX(float* X, float* nChild, float* Ancestor, float* Childs, float* indiceBusBegin, float* indiceChildBegin, float* nAgentByBus, float* CoresBusAgent, float* PosAgent, int Losstype, int nBus, int nAgent) {
	int bus = blockIdx.x;
	int index = threadIdx.x;
	int step = blockDim.x;

	int indice = indiceBusBegin[bus];
	if (bus < nBus) {
		int indiceChild = (bus < (nBus - 1)) ? indiceChildBegin[bus] : 0;
		int nb = nChild[bus];
		int nAgent = nAgentByBus[bus];

		if (index == 0 && bus > 0) { // Vai
			int Ai = Ancestor[bus];
			int indiceAi = indiceBusBegin[Ai];
			X[indice + 4 + 2 * nAgent] = X[indiceAi + 3];
		}


		for (int voisin = index; voisin < nb; voisin += step) { //  coalescent en ecriture mais pas en lecture 
			int c = Childs[indiceChild + voisin];
			int indiceBusChild = indiceBusBegin[c];
			X[indice + 5 + 2 * nAgent + voisin] = X[indiceBusChild];
			X[indice + 5 + 2 * nAgent + nb + voisin] = X[indiceBusChild + 1];
			X[indice + 5 + 2 * nAgent + 2 * nb + voisin] = X[indiceBusChild + 2];
		}
	}
	else {
		if (Losstype == 1) {// Current
			for (int bus2 = index; bus2 < nBus; bus2 += step) {
				int indicebus = indiceBusBegin[bus2];
				X[indice + 2 + bus2] = X[indicebus + 2];
			}
		}
		else { // Puissance
			for (int n = 1 + index; n < nAgent; n += step) {
				int bus2 = CoresBusAgent[n];
				int In = PosAgent[n];
				int nAgentBus = nAgentByBus[bus2];

				X[indice + n] = X[bus + 4 + In];
				X[indice + n + nAgent] = X[bus + 4 + nAgentBus + In];
			}/**/
		}
	}
}


__global__ void setPnFromX(float* Pn, float* X, float* indiceBusBegin, float* CoresAgentBus, float* nAgentByBus, float* beginBusAgent, int nAgent) {
	int bus = blockIdx.x;
	int thI = threadIdx.x;
	int step = blockDim.x;
	int begin = beginBusAgent[bus];
	int nB = nAgentByBus[bus];
	int fin = begin + nB;
	int indiceBus = indiceBusBegin[bus];


	for (int i = begin + thI; i < fin; i += step) { // ecriture coalecente mais pas lecture
		int agent = CoresAgentBus[i];

		Pn[agent] = X[indiceBus + 4 + thI];
		Pn[agent + nAgent] = X[indiceBus + 4 + nB + thI];

	}
}





// FIN OPF ADMM


// Debut PF

__global__ void calcWinterCar(float* Pinter, float* Qinter, float* VoltageCar, float* Glin, float* Blin, float* CoresVoiLin, float* CoresBusLin, float* nLines, int B) {


	int index = threadIdx.x;
	int step = blockDim.x;
	int i = blockIdx.x;
	extern __shared__ float shE[];
	int begin = CoresBusLin[i];
	int end = begin + nLines[blockIdx.x];
	int B2 = 2 * B;

	for (int n = index; n < B2; n += step)
	{
		shE[n] = VoltageCar[n];
	}
	__syncthreads();

	for (int l = begin + index; l < end; l += step) {
		int k = CoresVoiLin[l];
		float g = Glin[l];
		float b = Blin[l];

		float a = g * shE[k] - b * shE[k + B];
		float c = b * shE[k] + g * shE[k + B];

		Pinter[l] = shE[i] * a + shE[i + B] * c;
		Qinter[l] = shE[i + B] * a - shE[i] * c;

	}

}


__global__ void initECar(float* VoltageRealIm, float* E, int B) {


	int thIdx = threadIdx.x + blockDim.x * blockIdx.x;
	int size = gridDim.x * blockDim.x;
	for (int i = thIdx; i < B; i += size) {
		float V0 = E[i + B];
		float theta0 = E[i];
		VoltageRealIm[i] = V0 * cos(theta0);
		VoltageRealIm[i + B] = V0 * sin(theta0);
	}
}


__global__ void initECar(float* VoltageRealImD, float v0, float w0, int B) {


	int thIdx = threadIdx.x + blockDim.x * blockIdx.x;
	int size = gridDim.x * blockDim.x;
	for (int i = thIdx; i < B; i += size) {
		
		VoltageRealImD[i] = v0;
		VoltageRealImD[i + B] = w0;
	}
}

__global__ void calcEGPU(float* E, float* VoltageRealIm, int B) {

	int thIdx = threadIdx.x + blockDim.x * blockIdx.x;
	int size = gridDim.x * blockDim.x;

	for (int i = thIdx; i < B; i += size) {
		float Rev = VoltageRealIm[i];
		float Imv = VoltageRealIm[i + B];


		E[i + B] = sqrt(Rev * Rev + Imv * Imv);
		E[i] = atan2(Imv, Rev);
	}

}


__global__ void calculYGPU(float* Y, float* E, float* Voltage, float* Blin2, float* Glin2, float* CoresLineBus, int B, int L) {

	int thIdx = threadIdx.x + blockDim.x * blockIdx.x;
	int size = gridDim.x * blockDim.x;

	for (int b = thIdx; b < 2 * B; b += size) {
		Y[b] = E[b];
	}
	for (int l = thIdx; l < L; l += size) {
		int busFrom = CoresLineBus[l];
		int busTo = CoresLineBus[L + l];

		float ei = Voltage[busFrom];
		float fi = Voltage[busFrom + B];
		float ej = Voltage[busTo];
		float fj = Voltage[busTo + B];
		float Blin = Blin2[l];
		float Glin = Glin2[l];
		float Pij = (ei * ei + fi * fi - ei * ej - fi * fj) * Glin + (ei * fj - ej * fi) * Blin;
		Y[2 * B + l] = Pij;
	}

}


__global__ void calculYGPU(float* Y, float* E, float* Blin2, float* Glin2, float* CoresLineBus, int B, int L) {

	int thIdx = threadIdx.x + blockDim.x * blockIdx.x;
	int size = gridDim.x * blockDim.x;

	for (int b = thIdx; b < 2 * B; b += size) {
		Y[b] = E[b];
	}
	for (int l = thIdx; l < L; l += size) {
		int busFrom = CoresLineBus[l];
		int busTo = CoresLineBus[L + l];
		float vi = E[busFrom + B];
		float thetai = E[busFrom];
		float vj = E[busTo + B];
		float thetaj = E[busTo];

		float ei = vi * cos(thetai);
		float fi = vi * sin(thetai);
		float ej = vj * cos(thetaj);
		float fj = vj * sin(thetaj);
		float Blin = Blin2[l];
		float Glin = Glin2[l];
		float Pij = (ei * ei + fi * fi - ei * ej - fi * fj) * Glin + (ei * fj - ej * fi) * Blin;
		
		Y[2 * B + l] = Pij;
	}

}


__global__ void calculPhiGPU(float* Phi, float* E, float* Blin2, float* Glin2, float* CoresLineBus, int B, int L) {

	int thIdx = threadIdx.x + blockDim.x * blockIdx.x;
	int size = gridDim.x * blockDim.x;

	for (int l = thIdx; l < L; l += size) {
		int busFrom = CoresLineBus[l];
		int busTo = CoresLineBus[L + l];
		float vi = E[busFrom + B];
		float thetai = E[busFrom];
		float vj = E[busTo + B];
		float thetaj = E[busTo];

		float ei = vi * cos(thetai);
		float fi = vi * sin(thetai);
		float ej = vj * cos(thetaj);
		float fj = vj * sin(thetaj);
		float Blin = Blin2[l];
		float Glin = Glin2[l];
		float Pij = (ei * ei + fi * fi - ei * ej - fi * fj) * Glin + (ei * fj - ej * fi) * Blin;
		Phi[2 * B + l] = Pij;
	}

}



__global__ void setY(float* Y, float* E, float* Phi, int B, int L) {
	
	int thIdx = threadIdx.x + blockDim.x * blockIdx.x;
	int size = gridDim.x * blockDim.x;

	for (int b = thIdx; b < 2 * B; b += size) {
		Y[b] = E[b];
	}
	for (int l = thIdx; l < L; l += size) {
		Y[2 * B + l] = Phi[l];
	}
}
// Fin PF
/*

double b = coef[0];
double d = coef[1];
double e = coef[2];
int nRoot = 0;

if (b * b * b + 8 * d == 0) {
	//if (abs(b * b * b + 8 * d) < 0.00000001) {
	// passage de p^4 + b p^3 + d p + e -> a p^4 + b p^2 + c = 0
	double B = -3 * b * b / 8;
	double C = -3 * b * b * b * b / 256 - b * d / 4 + e;

	double delta = B * B - 4 * C;
	//std::cout << "Delta " << delta;
	if (delta == 0) {
		double z = -B / 2;
		nRoot = 2;
		//std::cout << " z " << z << std::endl;
		root[0] = sqrt(z);
		root[1] = -sqrt(z);
		return nRoot;
	}
	else if (delta > 0) {
		double z1 = (-B + sqrt(delta)) / 2;
		double z2 = (-B - sqrt(delta)) / 2;
		//std::cout << " z1 " << z1 << " z2 " << z2 << std::endl;
		if (z1 >= 0) {
			root[0] = sqrt(z1);
			root[1] = -sqrt(z1);
			nRoot = 2;
		} if (z2 >= 0) {
			root[nRoot] = sqrt(z2);
			root[nRoot + 1] = -sqrt(z2);
			nRoot += 2;
		}
		return nRoot;
	}
	else { // delta < 0
		//std::cout << "pas de racines réelle !!!! rip, on tente le pas bicarré" << std::endl;
	}
}

// for the lambda polynome
double coef2[2];
double rootlambda[3];
coef2[0] = (2 * b * d - 8 * e) / 8;
coef2[1] = -(b * b * e + d * d) / 8;
int nRootlambda = resolveRealPolynome3without2term(rootlambda, coef2);




for (int i = 0; i < nRootlambda; i++) {
	double lambda0 = rootlambda[i];
	//std::cout << "poly3 " << coef2[0] * lambda0 + coef2[1] + lambda0 * lambda0 * lambda0 << std::endl;
	double mu1 = 2 * lambda0 + (b * b) / 4;
	if (mu1 > 0) {
		double mu0 = sqrt(mu1);
		double DeltaP = -2 * lambda0 + 2 * (d - b * lambda0) / mu0 + b * mu0 + b * b / 2;
		double DeltaM = -2 * lambda0 - 2 * (d - b * lambda0) / mu0 - b * mu0 + b * b / 2;
		if (DeltaP >= 0) {
			root[nRoot] = (-mu0 + sqrt(DeltaP)) / 2 - b / 4;
			root[nRoot + 1] = (-mu0 - sqrt(DeltaP)) / 2 - b / 4;
			nRoot = nRoot + 2;
			//std::cout << "  Dp   ";
		}
		if (DeltaM >= 0) {
			root[nRoot] = (mu0 + sqrt(DeltaM)) / 2 - b / 4;
			root[nRoot + 1] = (mu0 - sqrt(DeltaM)) / 2 - b / 4;
			nRoot = nRoot + 2;
			//std::cout << "  DM   ";
		}
		if (nRoot > 0) {
			//std::cout << "poly4 " << coef[0] << " " <<  coef[1] << " " << coef[2] << std::endl;
			return nRoot;
		}
	}

}
double lambda0 = rootlambda[0];
double mu0 = sqrt(2 * lambda0 + (b * b) / 4);
double DeltaP = -2 * lambda0 + 2 * (d - b * lambda0) / mu0 + b * mu0 + b * b / 2;
double DeltaM = -2 * lambda0 - 2 * (d - b * lambda0) / mu0 - b * mu0 + b * b / 2;
std::cout << "no real root found " << abs(b * b * b + 8 * d) << " " << lambda0 << " " << mu0 << " " << DeltaP << " " << DeltaM << std::endl;



*/







