#include "hip/hip_runtime.h"
#include "../head/EndoPFGPU.cuh"
#define MAX(X, Y) X * (X >= Y) + Y * (Y > X)

// On prend la transpos�e de G !!! (ie G(n,i) = G[n*Nvar + i] )


EndoPFGPU::EndoPFGPU() : MethodP2P()
{
#if DEBUG_CONSTRUCTOR
	std::cout << " EndoPFGPU Constructor" << std::endl;
#endif // DEBUG_CONSTRUCTOR
	_name = NAME;
	timePerBlock = MatrixCPU(1, 9, 0); // Fb0, Fb1 , Fb2, Fb3, Fb5, Fb6 Fb0'
// si les sous ensemble ne sont pas accessible, tout est dans le premier.
	occurencePerBlock = MatrixCPU(1, 9, 0); //nb de fois utilis� pendant la simu
}


EndoPFGPU::EndoPFGPU(float rho) : MethodP2P()
{
#if DEBUG_CONSTRUCTOR
	std::cout << "default EndoPFGPU Constructor" << std::endl;
#endif // DEBUG_CONSTRUCTOR
	_name = NAME;
	_rho = rho;
	timePerBlock = MatrixCPU(1, 9, 0); // Fb0, Fb1 , Fb2, Fb3, Fb5, Fb6 Fb0'
// si les sous ensemble ne sont pas accessible, tout est dans le premier.
	occurencePerBlock = MatrixCPU(1, 9, 0); //nb de fois utilis� pendant la simu
}

EndoPFGPU::~EndoPFGPU()
{
	DELETEB(pf);
}
void EndoPFGPU::setParam(float rho)
{
	_rho = rho;
}

void EndoPFGPU::setTau(float tau)
{
	if (tau < 1) {
		throw std::invalid_argument("tau must be greater than 1");
	}
	_tau = tau;
}



void EndoPFGPU::solve(Simparam* result, const Simparam& sim, const StudyCase& cas)
{
#ifdef DEBUG_SOLVE
	cas.display();
	sim.display(1);
#endif // DEBUG_SOLVE
	
	timeEndoPF = clock();
#ifdef INSTRUMENTATION
	std::chrono::high_resolution_clock::time_point t1;
	std::chrono::high_resolution_clock::time_point t2;
#endif // INSTRUMENTATION


	// FB 0
	if (_id == 0) {
#ifdef INSTRUMENTATION
		t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION
		init(sim, cas);
#ifdef INSTRUMENTATION
		hipDeviceSynchronize();
		t2 = std::chrono::high_resolution_clock::now();
		timePerBlock.set(0, 0, (float) std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
		occurencePerBlock.set(0, 0, 1);
#endif // INSTRUMENTATION
	}
	_rhog = sim.getRho();
	_at1 = _rhog;
	
	int iterL = sim.getIterL();
	int stepL = sim.getStepL()/20;

	float epsL = sim.getEpsL();
	float epsG = sim.getEpsG();
	
	float resG = 2 * epsG;
	float epsL2 = epsL * epsL;
	_iterGlobal = 0;
	//CHECK_LAST_CUDA_ERROR();
	//Pn.display(true);
	//std::cout << "*******" << std::endl;
	while ((_iterGlobal < _iterG) && (resG>epsG) || (_iterGlobal <= _stepG)) {
		
#ifdef INSTRUMENTATION
		hipDeviceSynchronize();
		t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION
		updateLocalProbGPU(epsL2, iterL);
		//CHECK_LAST_CUDA_ERROR();
#ifdef INSTRUMENTATION
		hipDeviceSynchronize();
		t2 = std::chrono::high_resolution_clock::now();
		timePerBlock.increment(0, 1, (float) std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
#endif // INSTRUMENTATION
		
		tradeLin.swap(&Tlocal); // echange juste les pointeurs	


		updateGlobalProbGPU();
		//CHECK_LAST_CUDA_ERROR();
		if (!(_iterGlobal % _stepG)) {
#ifdef INSTRUMENTATION
			hipDeviceSynchronize();
			t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION
			resG = updateResBis(&resF, &Tlocal, _iterGlobal / _stepG, &tempNN);
			//CHECK_LAST_CUDA_ERROR();
#ifdef INSTRUMENTATION
			hipDeviceSynchronize();
			t2 = std::chrono::high_resolution_clock::now();
			timePerBlock.increment(0, 6, (float) std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
#endif // INSTRUMENTATION
		}
		_iterGlobal++;
	}
	//std::cout << _iterGlobal << " " << resG << " " << resF.get(0, (_iterGlobal - 1) / _stepG) << " " << resF.get(1, (_iterGlobal - 1) / _stepG) << " " << resF.get(2, (_iterGlobal - 1) / _stepG) << std::endl;
#ifdef INSTRUMENTATION
	occurencePerBlock.increment(0, 1, _iterGlobal);
	occurencePerBlock.increment(0, 3, _iterGlobal);
	occurencePerBlock.increment(0, 4, _iterGlobal);
	occurencePerBlock.increment(0, 5, _iterGlobal);
	occurencePerBlock.increment(0, 6, _iterGlobal / _stepG);

	hipDeviceSynchronize();
	t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION
	float fc = calcFc(&a, &b, &tradeLin, &Pn, &Ct, &tempN1, &tempNN);
	//Method::calcFc(MatrixGPU* cost1, MatrixGPU* cost2, MatrixGPU* trade, MatrixGPU* Pn, MatrixGPU* BETA, MatrixGPU* tempN1, MatrixGPU* tempNN)
	MatrixCPU tradeLinCPU;
	tradeLin.toMatCPU(tradeLinCPU);
	MatrixCPU LAMBDALinCPU;
	LAMBDALin.toMatCPU(LAMBDALinCPU);
	MatrixCPU PnCPU;
	Pn.toMatCPU(PnCPU);
	MatrixCPU MUCPU;
	MU.toMatCPU(MUCPU);
	//CHECK_LAST_CUDA_ERROR();
	int indice = 0;
	
	for (int idAgent = 0; idAgent < _nAgentTrue; idAgent++) {
		MatrixCPU omega(cas.getVoisin(idAgent));
		int Nvoisinmax = nVoisinCPU.get(idAgent, 0);
		for (int voisin = 0; voisin < Nvoisinmax; voisin++) {
			int idVoisin = omega.get(voisin, 0);
			trade.set(idAgent, idVoisin, tradeLinCPU.get(indice, 0));
			LAMBDA.set(idAgent, idVoisin, LAMBDALinCPU.get(indice, 0));
			indice = indice + 1;
		}
	}


	for (int idAgent = _nAgentTrue; idAgent < _nAgent; idAgent++) {
		for (int idVoisin = 0; idVoisin < _nAgentTrue; idVoisin++) {
			if (idVoisin != (idAgent - _nAgentTrue)) {
				trade.set(idAgent, idVoisin, tradeLinCPU.get(indice, 0));
				LAMBDA.set(idAgent, idVoisin, LAMBDALinCPU.get(indice, 0));
				indice = indice + 1;
			}

		}
	}
	


	// FB 5
	result->setResF(&resF);
	result->setLAMBDA(&LAMBDA);
	result->setTrade(&trade);
	//result->setDelta1(&delta1);
	//result->setDelta2(&delta2);
	result->setIter(_iterGlobal);
	
	result->setPn(&PnCPU);
	result->setFc(fc);
	result->setMU(&MUCPU);

#ifdef INSTRUMENTATION
	t2 = std::chrono::high_resolution_clock::now();
	timePerBlock.increment(0, 7, (float) std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
	occurencePerBlock.increment(0, 7, 1);
	result->setTimeBloc(&timePerBlock, &occurencePerBlock);
#endif // INSTRUMENTATION
	timeEndoPF = clock() - timeEndoPF;
	result->setTime((float)timeEndoPF / CLOCKS_PER_SEC);
	
}

void EndoPFGPU::updateP0(const StudyCase& cas)
{
	_id = _id + 1;
#ifdef INSTRUMENTATION
	std::chrono::high_resolution_clock::time_point t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION

	matLb.transferCPU();
	Pmin = MatrixGPU(cas.getPmin());
	Pmax = MatrixGPU(cas.getPmax());


	MatrixCPU Lb(cas.getLb());

	b = cas.getb();
	Cp1 = b;
	int indice = 0;

	for (int idAgent = 0; idAgent < _nAgentTrue; idAgent++) {
		int Nvoisinmax = nVoisinCPU.get(idAgent, 0);
		for (int voisin = 0; voisin < Nvoisinmax; voisin++) {
			matLb.set(indice, 0, Lb.get(idAgent, 0));
			indice = indice + 1;
		}
	}
	for (int idAgent = _nAgentTrue; idAgent < _nAgent; idAgent++) {
		for (int voisin = 0; voisin < (_nAgentTrue - 1); voisin++) {
			matLb.set(indice, 0, Lb.get(idAgent, 0));
			indice = indice + 1;
		}
	}

	matLb.transferGPU();

	Pmin.divideT(&nVoisin);
	Pmax.divideT(&nVoisin);
	Cp1.multiplyT(&nVoisin);
	

#ifdef INSTRUMENTATION
	hipDeviceSynchronize();
	std::chrono::high_resolution_clock::time_point t2 = std::chrono::high_resolution_clock::now();
	timePerBlock.increment(0, 8, (float) std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
	occurencePerBlock.increment(0, 8, 1);
#endif // INSTRUMENTATION


}

void EndoPFGPU::init(const Simparam& sim, const StudyCase& cas)
{
	DELETEB(pf);
	if (CoresMatLin.getPos()) { 
		CoresMatLin.transferCPU();
		CoresLinAgent.transferCPU();
		CoresAgentLin.transferCPU();
		CoresLinVoisin.transferCPU();
		CoresLinTrans.transferCPU();

		Tlocal_pre.transferCPU();
		tradeLin.transferCPU();
		LAMBDALin.transferCPU();

		matLb.transferCPU();
		matUb.transferCPU();
		Ct.transferCPU();
	}

	// intitilisation des matrixs et variables 
	
	//std::cout << "init " << std::endl;
	if (!cas.isAC()) {
		throw std::invalid_argument("Wrong studyCase must be AC");
	}

	_rhog = sim.getRho();
	_rho1 = sim.getRho1();
	_iterG = sim.getIterG();
	_stepG = sim.getStepG();
	float epsG = sim.getEpsG();
	float epsGC = sim.getEpsGC();
	_ratioEps = epsG / epsGC;
	isRadial = cas.isRadial();
	_nAgentTrue = sim.getNAgent();
	_nAgent = _nAgentTrue + _nAgentTrue;

	_rhol = _rho; //*nAgent
	//std::cout << "rho " << _rho << std::endl;
	if (_rho == 0) {
		_rhol = _rhog;
	}
	
	nVoisinCPU = cas.getNvoi();
	nVoisin = MatrixGPU(nVoisinCPU, 1);
	nVoisin.preallocateReduction();
	int nVoisinMax = nVoisin.max2();
	if (_blockSize * NMAXPEERPERTRHREAD < nVoisinMax) {
		std::cout << _blockSize << " " << NMAXPEERPERTRHREAD << " " << nVoisinMax << std::endl;
		throw std::invalid_argument("For this Method, an agent must not have more than 5120 peers");
	}

	//CHECK_LAST_CUDA_ERROR();
	_nLine = cas.getNLine(true);
	//std::cout << "_nLine " << _nLine << std::endl;
	_nBus = cas.getNBus();
	_nVarPF = _nLine + 2 * _nBus;

	//std::cout << _nVarPF << std::endl;
	
	_nTrade = nVoisin.sum();
	_nTradeP = nVoisin.sum(0,_nAgentTrue);
	//std::cout << "nTrade " << _nTrade << " " << _nTradeP << std::endl;
	//CHECK_LAST_CUDA_ERROR();
	_nTradeQ = _nTrade - _nTradeP;
	//std::cout << "nTrade " << _nTradeQ << " " << nVoisin.sum(_nAgentTrue, _nAgent) << std::endl;
	if (_nTradeQ != (_nAgentTrue * (_nAgentTrue - 1))) {
		std::cout << "err EndoPFGPU : " << _nAgent << " " << _nAgentTrue << " " << _nTrade << " " << _nTradeP << " " << _nTradeQ << std::endl;
		throw std::invalid_argument("Agent must be fully conected for the Q echanges, WIP");
	}
	_numBlocksN = ceil((_nAgent + _blockSize - 1) / _blockSize);
	_numBlocksM = ceil((_nTrade + _blockSize - 1) / _blockSize);
	_numBlocksL = ceil((_nLine + _blockSize - 1) / _blockSize);
	_numBlocksBL = ceil((_nVarPF + _blockSize - 1) / _blockSize);
	
	
	//std::cout << _numBlocksN << " " << _numBlocksM << " " << _numBlocksL << " " << _numBlocksBL << std::endl;
	
	//std::cout <<  _blockSize << std::endl;
	if (initWithMarketClear) {
		ADMMMarketGPU market;
		Simparam res(sim);
		market.solve(&res, sim, cas);
		//res.display();
		LAMBDA = res.getLambda();
		trade = res.getTrade();
		Pnpre = MatrixGPU(res.getPn(), 1);

	}
	else {
		LAMBDA = sim.getLambda();
		trade = sim.getTrade();
		Pnpre = MatrixGPU(sim.getPn(), 1);
	}
	//Pnpre.display(true);
	Tmoy = Pnpre;
	Tmoy.divideT(&nVoisin);
	//std::cout << "*******" << std::endl;
	
	
	_at1 = _rhog; // represente en fait 2*a
	_at2 = _rhol;

	resF = MatrixCPU(3, (_iterG / _stepG) + 1);
	resX = MatrixCPU(4, (_iterG / _stepG) + 1);

	MatrixGPU BETA(cas.getBeta());
	
	MatrixGPU Ub(cas.getUb());
	MatrixGPU Lb(cas.getLb());
	
	 
	//std::cout << "mise sous forme lin�aire" << std::endl;
	


	CoresMatLin = MatrixGPU(_nAgent, _nAgentTrue, -1);
	CoresLinAgent = MatrixGPU(_nTrade, 1);
	CoresAgentLin = MatrixGPU(_nAgent + 1, 1);
	CoresLinVoisin = MatrixGPU(_nTrade, 1);
	CoresLinTrans = MatrixGPU(_nTrade, 1);

	Tlocal_pre = MatrixGPU(_nTrade, 1);
	tradeLin = MatrixGPU(_nTrade, 1);
	LAMBDALin = MatrixGPU(_nTrade, 1);

	matLb = MatrixGPU(_nTrade, 1);
	matUb = MatrixGPU(_nTrade, 1);
	Ct = MatrixGPU(_nTrade, 1);
	

	int indice = 0;
	//std::cout << " P " << std::endl;
	for (int idAgent = 0; idAgent < _nAgentTrue; idAgent++) { // P
		MatrixGPU omega(cas.getVoisin(idAgent));
		int Nvoisinmax = nVoisinCPU.get(idAgent, 0);
		for (int voisin = 0; voisin < Nvoisinmax; voisin++) {
			int idVoisin = omega.get(voisin, 0);
			if(Lb.getNCol()== 1){
				matLb.set(indice, 0, Lb.get(idAgent, 0));
				matUb.set(indice, 0, Ub.get(idAgent, 0));
			} else {
				matLb.set(indice, 0, Lb.get(idAgent, idVoisin));
				matUb.set(indice, 0, Ub.get(idAgent, idVoisin));
			}
			Ct.set(indice, 0, BETA.get(idAgent, idVoisin));
			tradeLin.set(indice, 0, trade.get(idAgent, idVoisin));
			Tlocal_pre.set(indice, 0, trade.get(idAgent, idVoisin));
			LAMBDALin.set(indice, 0, LAMBDA.get(idAgent, idVoisin));
			CoresLinAgent.set(indice, 0, idAgent);
			CoresLinVoisin.set(indice, 0, idVoisin);
			CoresMatLin.set(idAgent, idVoisin, indice);
			indice = indice + 1;
		}
		CoresAgentLin.set(idAgent + 1, 0, indice);
	}
	//std::cout << " Q " << std::endl;
	for (int idAgent = _nAgentTrue; idAgent < _nAgent; idAgent++) { // Q
		for (int idVoisin = 0; idVoisin < _nAgentTrue; idVoisin++) {
			if (idVoisin != (idAgent - _nAgentTrue)) {
				matLb.set(indice, 0, Lb.get(idAgent, 0));
				matUb.set(indice, 0, Ub.get(idAgent, 0));
				//Ct.set(indice, 0, BETA.get(idAgent, idVoisin));
				tradeLin.set(indice, 0, trade.get(idAgent, idVoisin));
				Tlocal_pre.set(indice, 0, trade.get(idAgent, idVoisin));
				LAMBDALin.set(indice, 0, LAMBDA.get(idAgent, idVoisin));
				CoresLinAgent.set(indice, 0, idAgent);
				CoresLinVoisin.set(indice, 0, idVoisin + _nAgentTrue);
				CoresMatLin.set(idAgent, idVoisin, indice);
				indice = indice + 1;
			}
		}
		CoresAgentLin.set(idAgent + 1, 0, indice);
	}
	for (int lin = 0; lin < _nTrade; lin++) {
		int i = CoresLinAgent.get(lin, 0);
		int j = CoresLinVoisin.get(lin, 0);
		if (lin >= _nTradeP) {
			i -= _nAgentTrue;
		}

		int k = CoresMatLin.get(j, i);
		CoresLinTrans.set(lin, 0, k);
	}
	// transfert des mises lineaire
	matUb.transferGPU();
	matLb.transferGPU();
	Ct.transferGPU();

	Tlocal_pre.transferGPU();
	tradeLin.transferGPU();
	LAMBDALin.transferGPU();

	CoresAgentLin.transferGPU();
	CoresLinAgent.transferGPU();
	CoresLinVoisin.transferGPU();
	CoresMatLin.transferGPU();
	CoresLinTrans.transferGPU();

	
	//CHECK_LAST_CUDA_ERROR();
	//std::cout << "donnees sur CPU pour le grid" << std::endl;
	delta1 = MatrixGPU(_nVarPF, 1, 0, 1);
	delta2 = MatrixGPU(_nVarPF, 1, 0, 1);
	Z1 = MatrixGPU(_nVarPF, 1, 0, 1);
	Z2 = MatrixGPU(_nVarPF, 1, 0, 1);
	Y = MatrixGPU(_nVarPF, 1, 0, 1);
	Ypre = MatrixGPU(_nVarPF, 1, 0, 1);
	dY = MatrixGPU(_nVarPF, 1, 0, 1);

	Ylimit = MatrixGPU(_nVarPF, 1, 0, 1); // angle, amplitude, flux
	YOffset = MatrixGPU(_nVarPF, 1, 0, 1); // angle, amplitude, flux
	G = MatrixGPU(_nAgent, _nVarPF, 0, 1);
	SensiBis = MatrixGPU(_nVarPF, 1, 0, 1);
	
	
	MatrixGPU LimitsUb(cas.getUpperBound(), 1); // angle, amplitude, flux
	MatrixGPU LimitsLb(cas.getLowerBound(), 1);
	
	initLimits << <_numBlocksBL, _blockSize >> > (Ylimit._matrixGPU, YOffset._matrixGPU, LimitsLb._matrixGPU, LimitsUb._matrixGPU, _nVarPF);
	//Ylimit.display(true);
	//YOffset.display(true);
	
	CHECK_LAST_CUDA_ERROR();

	//std::cout << " PF " << std::endl;
	if (isRadial) {
		pf = new GPUPFdistPQ;
	}
	else {
		pf = new GPUPF;
	}
	//Ylimit.display();
	//YOffset.display();

	pf->init(cas, &Pnpre);

	
	//CHECK_LAST_CUDA_ERROR();
	//std::cout << "autres donn�e sur CPU" << std::endl;
	tempNN = MatrixGPU(_nTrade, 1, 0, 1);
	tempNN.preallocateReduction();
	tempN1 = MatrixGPU(_nAgent, 1, 0, 1); // plut�t que de re-allouer de la m�moire � chaque utilisation
	tempL1 = MatrixGPU(_nVarPF, 1, 0, 1);
	tempL1.preallocateReduction();
	//MatrixGPU temp1N(1, _nAgent, 0, 1);

	Tlocal = MatrixGPU(_nTrade, 1, 0, 1);
	Tlocal.preallocateReduction();
	

	P = MatrixGPU(_nAgent, 1, 0, 1); // moyenne des trades
	Pn = MatrixGPU(sim.getPn(), 1);
	dP = MatrixGPU(_nAgent, 1, 0, 1);

	a = MatrixGPU(cas.geta(), 1);
	b = MatrixGPU(cas.getb(), 1);
	Ap2 = a;
	Ap1 = nVoisin;
	Ap12 = MatrixGPU(_nAgent, 1, 0, 1);

	Bt1 = MatrixGPU(_nTrade, 1, 0, 1);
	Bt2 = MatrixGPU(_nTrade, 1, 0, 1);
	Cp = MatrixGPU(_nAgent, 1, 0, 1);
	Cp2 = MatrixGPU(_nAgent, 1, 0, 1);
	Cp1 = b;
	Bp1 = MatrixGPU(_nAgent, 1, 0, 1);

	Pmin = MatrixGPU(cas.getPmin(), 1);
	Pmax = MatrixGPU(cas.getPmax(), 1);
	

	MU = MatrixGPU(sim.getMU(), 1); // facteur reduit i.e lambda_l/_rho
	

	Pmin.divideT(&nVoisin);
	Pmax.divideT(&nVoisin);
	
	


	Ap1.multiply(_rhol);
	Ap2.multiplyT(&nVoisin);
	Ap2.multiplyT(&nVoisin);
	Ap12.add(&Ap1, &Ap2);

	Cp1.multiplyT(&nVoisin);
	
	
	

	//std::cout << "update Global" << std::endl;
	updateGlobalProbGPU();
	//std::cout << "rho " << _rhog << " rhoL " << _rhol << " rho1 " << _rho1 << std::endl;
	//std::cout << "fin init temps : " << (float)(clock() - t) / CLOCKS_PER_SEC << std::endl;
	//std::cout << "fin init " << std::endl;
	CHECK_LAST_CUDA_ERROR();

}

void EndoPFGPU::updateGlobalProbGPU() {
	
	// FB 2a
#ifdef INSTRUMENTATION
	hipDeviceSynchronize();
	std::chrono::high_resolution_clock::time_point t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION
	//std::cout << "*************" << std::endl;
	
	Pn.swap(&Pnpre);
	updatePn(&Pn,&Tmoy,&nVoisin);
	//std::cout << " Tmoy " << std::endl;
	
	//std::cout << " Pn " << std::endl;
	//Pn.display(true);
//	tradeLin.display(true);
	
	//std::cout << "update PF" << std::endl;
	pf->updatePQ(&Pn);
	
#ifdef INSTRUMENTATION
	hipDeviceSynchronize();
	std::chrono::high_resolution_clock::time_point t2 = std::chrono::high_resolution_clock::now();

	timePerBlock.increment(0, 3, (float) std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());

	// FB 3b
	hipDeviceSynchronize();
	t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION	
	 

	//std::cout << "solve PF" << std::endl;
	
	// FB 3
	pf->solve();
	 

	tempL1 = pf->getY();
	Ypre.swap(&Y);
	Y.subtract(&tempL1, &YOffset);
	
#ifdef INSTRUMENTATION
	hipDeviceSynchronize();
	t2 = std::chrono::high_resolution_clock::now();
	timePerBlock.increment(0, 4, (float) std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());

	// FB 3c
	hipDeviceSynchronize();
	t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION
	//std::cout << "update PF" << std::endl;
	//std::cout << "Y " << std::endl;
	//Y.display(true);
	
	float Ploss = - pf->getPloss() ;
	float Qloss = - pf->getQloss();	
	//std::cout << " Ploss " << Ploss << " Qloss " << Qloss << std::endl;
	
	Pmin.set(0, 0, Ploss / nVoisinCPU.get(0, 0), true);
	Pmax.set(0, 0, Ploss / nVoisinCPU.get(0, 0), true);
	
	Pmin.set(_nAgentTrue, 0, Qloss / nVoisinCPU.get(_nAgentTrue, 0), true);
	Pmax.set(_nAgentTrue, 0, Qloss / nVoisinCPU.get(_nAgentTrue, 0), true);
	
	if (Qloss > 0) {
		matUb.setBloc(_nTradeP, _nTradeP + _nAgentTrue - 1, 0, 1, Qloss);
	}
	else {
		matLb.setBloc(_nTradeP, _nTradeP + _nAgentTrue - 1, 0, 1, Qloss);
	}/**/
	
 
	


	updateZDeltaGPU << <_numBlocksBL, _blockSize >> > (Z1._matrixGPU, Z2._matrixGPU, Ylimit._matrixGPU, delta1._matrixGPU, delta2._matrixGPU, Y._matrixGPU, _nVarPF);

	updateSensi();
	
	updateCp2();
	Cp.add(&Cp1, &Cp2);
/*std::cout << " Z " << std::endl;
	Z1.display(true);
	Z2.display(true);

	std::cout << " Delta " << std::endl;
	delta1.display(true);
	delta2.display(true);
std::cout << " Sensi " << std::endl;
	SensiBis.display(true);
	G.display(true);	
	std::cout << " Cp " << std::endl;
	Cp2.display(true);
	//Cp.display(true);
*/
	////CHECK_LAST_CUDA_ERROR();

	

	// FB 3c
	updateLAMBDABt1GPU << <_numBlocksM, _blockSize >> > (Bt1._matrixGPU, LAMBDALin._matrixGPU, tradeLin._matrixGPU, _rhog, CoresLinTrans._matrixGPU, _nTrade);
	 
	
#ifdef INSTRUMENTATION
	t2 = std::chrono::high_resolution_clock::now();
	timePerBlock.increment(0, 5, (float) std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
#endif // INSTRUMENTATION
	
}

void EndoPFGPU::updateLocalProbGPU(float epsL, int nIterL) {
	// FB 1a
	int numBlocks = _nAgent;
	/*std::cout << "probl�me local" << std::endl;
	std::cout << _at1 << " " << _at2 << std::endl;
	Bt1.display(true);
	Ct.display(true);
	matLb.display(true);
	matUb.display(true);
	Ap1.display(true);
	Ap12.display(true);
	Bp1.display(true);
	Cp.display(true);
	Pmin.display(true);
	Pmax.display(true);*/


	switch (_blockSize) {
	case 512:
		updateTradePGPUSharedResidual<512> << <numBlocks, _blockSize >> > (Tlocal._matrixGPU, Tlocal_pre._matrixGPU, Tmoy._matrixGPU, P._matrixGPU, MU._matrixGPU, nVoisin._matrixGPU, _at1, _at2, Bt1._matrixGPU, Ct._matrixGPU,
			matLb._matrixGPU, matUb._matrixGPU, Ap1._matrixGPU, Ap12._matrixGPU, Cp._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, CoresAgentLin._matrixGPU, epsL, nIterL);
		break;
	case 256:
		updateTradePGPUSharedResidual<256> << <numBlocks, _blockSize >> > (Tlocal._matrixGPU, Tlocal_pre._matrixGPU, Tmoy._matrixGPU, P._matrixGPU, MU._matrixGPU, nVoisin._matrixGPU, _at1, _at2, Bt1._matrixGPU, Ct._matrixGPU,
			matLb._matrixGPU, matUb._matrixGPU, Ap1._matrixGPU, Ap12._matrixGPU, Cp._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, CoresAgentLin._matrixGPU, epsL, nIterL);
		break;
	case 128:
		updateTradePGPUSharedResidual<128> << <numBlocks, _blockSize >> > (Tlocal._matrixGPU, Tlocal_pre._matrixGPU, Tmoy._matrixGPU, P._matrixGPU, MU._matrixGPU, nVoisin._matrixGPU, _at1, _at2, Bt1._matrixGPU, Ct._matrixGPU,
			matLb._matrixGPU, matUb._matrixGPU, Ap1._matrixGPU, Ap12._matrixGPU, Cp._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, CoresAgentLin._matrixGPU, epsL, nIterL);
		break;
	case 64:
		updateTradePGPUSharedResidual< 64> << <numBlocks, _blockSize >> > (Tlocal._matrixGPU, Tlocal_pre._matrixGPU, Tmoy._matrixGPU, P._matrixGPU, MU._matrixGPU, nVoisin._matrixGPU, _at1, _at2, Bt1._matrixGPU, Ct._matrixGPU,
			matLb._matrixGPU, matUb._matrixGPU, Ap1._matrixGPU, Ap12._matrixGPU, Cp._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, CoresAgentLin._matrixGPU, epsL, nIterL);
		break;
	case 32:
		updateTradePGPUSharedResidual< 32> << <numBlocks, _blockSize >> > (Tlocal._matrixGPU, Tlocal_pre._matrixGPU, Tmoy._matrixGPU, P._matrixGPU, MU._matrixGPU, nVoisin._matrixGPU, _at1, _at2, Bt1._matrixGPU, Ct._matrixGPU,
			matLb._matrixGPU, matUb._matrixGPU, Ap1._matrixGPU, Ap12._matrixGPU, Cp._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, CoresAgentLin._matrixGPU, epsL, nIterL);
		break;
	case 16:
		updateTradePGPUSharedResidual< 16> << <numBlocks, _blockSize >> > (Tlocal._matrixGPU, Tlocal_pre._matrixGPU, Tmoy._matrixGPU, P._matrixGPU, MU._matrixGPU, nVoisin._matrixGPU, _at1, _at2, Bt1._matrixGPU, Ct._matrixGPU,
			matLb._matrixGPU, matUb._matrixGPU, Ap1._matrixGPU, Ap12._matrixGPU, Cp._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, CoresAgentLin._matrixGPU, epsL, nIterL);
		break;
	case  8:
		updateTradePGPUSharedResidual<  8> << <numBlocks, _blockSize >> > (Tlocal._matrixGPU, Tlocal_pre._matrixGPU, Tmoy._matrixGPU, P._matrixGPU, MU._matrixGPU, nVoisin._matrixGPU, _at1, _at2, Bt1._matrixGPU, Ct._matrixGPU,
			matLb._matrixGPU, matUb._matrixGPU, Ap1._matrixGPU, Ap12._matrixGPU, Cp._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, CoresAgentLin._matrixGPU, epsL, nIterL);
		break;
	case  4:
		updateTradePGPUSharedResidual<  4> << <numBlocks, _blockSize >> > (Tlocal._matrixGPU, Tlocal_pre._matrixGPU, Tmoy._matrixGPU, P._matrixGPU, MU._matrixGPU, nVoisin._matrixGPU, _at1, _at2, Bt1._matrixGPU, Ct._matrixGPU,
			matLb._matrixGPU, matUb._matrixGPU, Ap1._matrixGPU, Ap12._matrixGPU, Cp._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, CoresAgentLin._matrixGPU, epsL, nIterL);
		break;
	case  2:
		updateTradePGPUSharedResidual<  2> << <numBlocks, _blockSize >> > (Tlocal._matrixGPU, Tlocal_pre._matrixGPU, Tmoy._matrixGPU, P._matrixGPU, MU._matrixGPU, nVoisin._matrixGPU, _at1, _at2, Bt1._matrixGPU, Ct._matrixGPU,
			matLb._matrixGPU, matUb._matrixGPU, Ap1._matrixGPU, Ap12._matrixGPU, Cp._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, CoresAgentLin._matrixGPU, epsL, nIterL);
		break;
	case  1:
		updateTradePGPUSharedResidual<  1> << <numBlocks, _blockSize >> > (Tlocal._matrixGPU, Tlocal_pre._matrixGPU, Tmoy._matrixGPU, P._matrixGPU, MU._matrixGPU, nVoisin._matrixGPU, _at1, _at2, Bt1._matrixGPU, Ct._matrixGPU,
			matLb._matrixGPU, matUb._matrixGPU, Ap1._matrixGPU, Ap12._matrixGPU, Cp._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, CoresAgentLin._matrixGPU, epsL, nIterL);
		break;
	}

	

}


void EndoPFGPU::updateCp2()
{
	int numBlocks = _nAgent;
	switch (_blockSize) {
	case 512:
		updateCp2GPU<512> << <numBlocks, _blockSize >> > (Cp2._matrixGPU, SensiBis._matrixGPU, G._matrixGPU, nVoisin._matrixGPU, _rho1, _nVarPF);
		break;
	case 256:
		updateCp2GPU<256> << <numBlocks, _blockSize >> > (Cp2._matrixGPU, SensiBis._matrixGPU, G._matrixGPU, nVoisin._matrixGPU, _rho1, _nVarPF);
		break;
	case 128:
		updateCp2GPU<128> << <numBlocks, _blockSize >> > (Cp2._matrixGPU, SensiBis._matrixGPU, G._matrixGPU, nVoisin._matrixGPU, _rho1, _nVarPF);
		break;
	case 64:
		updateCp2GPU< 64> << <numBlocks, _blockSize >> > (Cp2._matrixGPU, SensiBis._matrixGPU, G._matrixGPU, nVoisin._matrixGPU, _rho1, _nVarPF);
		break;
	case 32:
		updateCp2GPU< 32> << <numBlocks, _blockSize >> > (Cp2._matrixGPU, SensiBis._matrixGPU, G._matrixGPU, nVoisin._matrixGPU, _rho1, _nVarPF);
		break;
	case 16:
		updateCp2GPU< 16> << <numBlocks, _blockSize >> > (Cp2._matrixGPU, SensiBis._matrixGPU, G._matrixGPU, nVoisin._matrixGPU, _rho1, _nVarPF);
		break;
	case  8:
		updateCp2GPU<  8> << <numBlocks, _blockSize >> > (Cp2._matrixGPU, SensiBis._matrixGPU, G._matrixGPU, nVoisin._matrixGPU, _rho1, _nVarPF);
		break;
	case  4:
		updateCp2GPU<  4> << <numBlocks, _blockSize >> > (Cp2._matrixGPU, SensiBis._matrixGPU, G._matrixGPU, nVoisin._matrixGPU, _rho1, _nVarPF);
		break;
	case  2:
		updateCp2GPU<  2> << <numBlocks, _blockSize >> > (Cp2._matrixGPU, SensiBis._matrixGPU, G._matrixGPU, nVoisin._matrixGPU, _rho1, _nVarPF);
		break;
	case  1:
		updateCp2GPU<  1> << <numBlocks, _blockSize >> > (Cp2._matrixGPU, SensiBis._matrixGPU, G._matrixGPU, nVoisin._matrixGPU, _rho1, _nVarPF);
		break;
	}



}



void EndoPFGPU::updateSensi()
{

	updateSensiBis<<<_numBlocksBL,_blockSize>>> (SensiBis._matrixGPU, Y._matrixGPU, Z1._matrixGPU, Z2._matrixGPU, delta1._matrixGPU, delta2._matrixGPU, _nVarPF);
	dY.subtract(&Y, &Ypre);
	dP.subtract(&Pn, &Pnpre);
	//dY.display(true);
	//dP.display(true);
	updateSensiGPU << <_nAgent, _blockSize >> > (G._matrixGPU, dY._matrixGPU, dP._matrixGPU, _nVarPF);
	//G.display();
}




float EndoPFGPU::updateResBis(MatrixCPU* res, MatrixGPU* Tlocal, int iter, MatrixGPU* tempNN)
{
	
	float resS = Tlocal->max2(&tradeLin);

	updateDiffGPU << <_numBlocksM, _blockSize >> > (tempNN->_matrixGPU, tradeLin._matrixGPU, CoresLinTrans._matrixGPU, _nTrade);
	float resR = tempNN->max2();
	// Residus reseau
	
	tempL1.subtract(&Ylimit, &Y);
	tempL1.projectNeg();

	float resXf = _ratioEps * tempL1.max2();
	res->set(0, iter, resR);
	res->set(1, iter, resS);
	res->set(2, iter, resXf);
	return MAX(MAX(resXf, resS), resR);
}

void EndoPFGPU::display() {

	a.transferCPU();
	b.transferCPU();
	Pn.transferCPU();
	Pmin.transferCPU();
	Pmax.transferCPU();

	if (_iterGlobal == 0) {
		std::cout << "algorithm not launch" << std::endl;
	}
	else if (_iterGlobal < _iterG) {
		std::cout << "method " << _name << " converged in " << _iterGlobal << " iterations." << std::endl;
		std::cout << "Converged in " << (float) timeEndoPF / CLOCKS_PER_SEC << " seconds" << std::endl;

	}
	else {
		std::cout << "method " << _name << " not converged in " << _iterGlobal << " iterations." << std::endl;
		std::cout << "time taken " << (float) timeEndoPF / CLOCKS_PER_SEC << " seconds" << std::endl;
	}
	std::cout << "The power error of this state is (constraint) " << resF.get(0, _iterGlobal / _stepG) << " and convergence " << resF.get(1, _iterGlobal / _stepG) << std::endl;
	std::cout << "===============================================================|" << std::endl;
	std::cout << "      System Summary                                           |" << std::endl;
	std::cout << "===============================================================|" << std::endl;
	std::cout << "Agent            " << _nAgentTrue << std::endl;



	std::cout << std::endl << std::endl;
	std::cout << std::endl << std::endl;
	std::cout << "========================================================================================================|" << std::endl;
	std::cout << "      Agent Data                                                                                        |" << std::endl;
	std::cout << "========================================================================================================|" << std::endl;
	std::cout << " Agent  |  Cost    |  Cost    |          Power Injection           |           Power Injection          |" << std::endl;
	std::cout << "  #     |   a (pu) |   b (pu) |  P (pu)  | Pmin (pu)  | Pmax (pu)  |  Q (pu)   | Qmin (pu)  | Qmax (pu) |" << std::endl;
	std::cout << "--------|----------|----------|----------|------------|------------|-----------|------------|-----------|" << std::endl;

	for (int n = 0; n < _nAgentTrue; n++) {

		std::cout << std::setw(8) << n << "|" << std::setw(9) << a.get(n, 0) << " |" << std::setw(10)
			<< b.get(n, 0) << "|" << std::setw(10) << Pn.get(n, 0) << "|" << std::setw(12)
			<< Pmin.get(n, 0) * nVoisinCPU.get(n, 0) << "|" << std::setw(12) << Pmax.get(n, 0) * nVoisinCPU.get(n, 0)
			<< "|" << std::setw(11) << Pn.get(n + _nAgentTrue, 0) << "|" << std::setw(12) << Pmin.get(n + _nAgentTrue, 0) * nVoisinCPU.get(n + _nAgentTrue, 0)
			<< "|" << std::setw(11) << Pmax.get(n + _nAgentTrue, 0) * nVoisinCPU.get(n + _nAgentTrue, 0) << "|" << std::endl;
	}


	std::cout << "========================================================================================================|" << std::endl;
	std::cout << "                      END PRINT                                                                         |" << std::endl;
	std::cout << "========================================================================================================|" << std::endl;


	a.transferGPU();
	b.transferGPU();
	Pn.transferGPU();
	Pmin.transferGPU();
	Pmax.transferGPU();
}





__global__ void initLimits(float* Ylimit, float* Yoffset, float* limitsLb, float* limitsUb, int nVarPF) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int step = blockDim.x * gridDim.x;

	for (int i = index; i < nVarPF; i += step) {
		float ub = limitsUb[i];
		float lb = limitsLb[i];
		float mid = (ub + lb) / 2;
		float lim = ub - mid;
		Ylimit[i] = lim;
		Yoffset[i] = mid;
	}
}



/*
	Z1.add(&Ylimit, &delta1);
	Z1.subtract(&Y);
	Z1.projectPos();

	Z2.add(&Ylimit, &delta2);
	Z2.add(&Y);
	Z2.projectPos();
*/

__global__ void updateZGPU(float* Z1, float* Z2, float* Ylimit, float* delta1, float* delta2, float* Y, int nVarPF) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int step = blockDim.x * gridDim.x;

	for (int i = index; i < nVarPF; i += step) {

		float z1 = Ylimit[i] + delta1[i] - Y[i];
		float z2 = Ylimit[i] + delta2[i] + Y[i];
		Z1[i] = (z1 > 0) * z1;
		Z2[i] = (z2 > 0) * z2;

	}
}


/*
	delta1.add(&Ylimit);
	delta1.subtract(&Z1);
	delta1.subtract(&Y);

	delta2.add(&Ylimit);
	delta2.subtract(&Z2);
	delta2.add(&Y);
*/
__global__ void updateDeltaGPU(float* delta1, float* delta2, float* Z1, float* Z2, float* Ylimit, float* Y, int nVarPF) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int step = blockDim.x * gridDim.x;

	for (int i = index; i < nVarPF; i += step) {
		float d1 = Ylimit[i] + delta1[i] - Y[i];
		float d2 = Ylimit[i] + delta2[i] + Y[i];

		delta1[i] = (d1 < 0) * d1;//delta1[i] + Ylimit[i] - Z1[i] - Y[i];
		delta2[i] = (d2 < 0) * d2;//delta2[i] + Ylimit[i] - Z2[i] + Y[i];
	}
}


__global__ void updateZDeltaGPU(float* Z1, float* Z2, float* Ylimit, float* delta1, float* delta2, float* Y, int nVarPF) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int step = blockDim.x * gridDim.x;

	for (int i = index; i < nVarPF; i += step) {

		float z1 = Ylimit[i] + delta1[i] - Y[i];
		float z2 = Ylimit[i] + delta2[i] + Y[i];
		Z1[i] = (z1 > 0) * z1;
		Z2[i] = (z2 > 0) * z2;
		delta1[i] = (z1 < 0) * z1;
		delta2[i] = (z2 < 0) * z2;
	}

}



/*
for (int n = 0; n < _nAgent; n++) {
		float sum = 0;
		for (int i = 0; i < _nVarPF; i++) {
			sum += SensiBis.get(i, 0) * G.get(i, n);
		}
		Cp2.set(n, 0, sum * _rho1 * nVoisin.get(n, 0));
	}

*/

template <unsigned int blockSize>
__global__ void updateCp2GPU(float* Cp2, float* SensiBis, float* G, float* nVoisin, float rho1, int nVarPF) {
	int index = threadIdx.x;
	int step = blockDim.x;
	int agent = blockIdx.x; // un bloc par agent
	__shared__ float shArr[blockSize];

	float sum = 0;
	for (int i = index; i < nVarPF; i += step) {
		sum += SensiBis[i] * G[agent * nVarPF + i];
	}
	shArr[index] = sum;
	__syncthreads();

	if (blockSize >= 512) { if (index < 256) { shArr[index] += shArr[index + 256]; } __syncthreads(); }
	if (blockSize >= 256) { if (index < 128) { shArr[index] += shArr[index + 128]; } __syncthreads(); }
	if (blockSize >= 128) { if (index < 64) { shArr[index] += shArr[index + 64]; } __syncthreads(); }
	if (index < 32) {
		warpReduce<blockSize>(shArr, index);
	}
	if (index == 0) {
		Cp2[agent] = shArr[0] * rho1 * nVoisin[agent];
	}

}


/*
	SensiBis.set(&Y);
	SensiBis.multiply(2);
	SensiBis.add(&Z1);
	SensiBis.subtract(&Z2);
	SensiBis.add(&delta2);
	SensiBis.subtract(&delta1);
*/

__global__ void updateSensiBis(float* sensiBis, float* Y, float* Z1, float* Z2, float* delta1, float* delta2, int nVarPF) {

	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int step = blockDim.x * gridDim.x;

	for (int i = index; i < nVarPF; i += step) {
		sensiBis[i] = 2 * Y[i] + Z1[i] - Z2[i] + delta2[i] - delta1[i];
	}

}



/*

	for (int i = 0; i < _nVarPF; i++) {
		for (int n = 1; n < _nAgent; n++) {
			if (abs(dP.get(n,0)) > 0.01) {
				G.set(i, n, dY.get(i, 0) / dP.get(n, 0));
			}
		}
	}

*/

__global__ void updateSensiGPU(float* G, float* dY, float* dP, int nVar) {
	int index = threadIdx.x;
	int step = blockDim.x;
	int agent = blockIdx.x; // un bloc par agent
	float dp = dP[agent];

	if (dp > 0.01 || dp < -0.01) {
		for (int i = index; i < nVar; i += step) {
			G[agent * nVar + i] = dY[i] / dp;
		}
	}



}







/*
template <unsigned int blockSize>
__global__ void updateTradePGPUSharedResidual(float* Tlocal, float* Tlocal_pre, float* Tmoy, float* P, float* MU, float* nVoisin, float at1, float at2, float* Bt1, float* Ct,
	float* matLb, float* matUb, float* Ap1, float* Ap12, float* Cp, float* Pmin, float* Pmax, float* CoresAgentLin, float eps, int nStepL) {

	//Definition de toutes les variables locales
	int i = blockIdx.x; // c'est aussi l'identifiant de l'agent !
	unsigned int thIdx = threadIdx.x;
	const int step = blockSize;
	// ne change pas


	float Bt1local[NMAXPEERPERTRHREAD];
	float Ctlocal[NMAXPEERPERTRHREAD];
	float matUblocal[NMAXPEERPERTRHREAD];
	float matLblocal[NMAXPEERPERTRHREAD];

	float Tlocallocal[NMAXPEERPERTRHREAD]; // change
	float Tlocalprelocal[NMAXPEERPERTRHREAD]; // change
	float sum;
	float bp, MULOCAL, moy, p;
	float m, r, ub, lb, t;
	// le changement doit �tre partag� par tous les threads du bloc

	__shared__ float MuShared;
	__shared__ float TMoyShared;
	__shared__ float PShared;


	// constant et commun � tous les thread d'un bloc
	__shared__ float Ap1Shared;
	__shared__ float CpShared;
	__shared__ float Ap12Shared;
	__shared__ float PmaxShared;
	__shared__ float PminShared;
	__shared__ float nVoisinShared;
	__shared__ float at1Shared;
	__shared__ float at2Shared;
	__shared__ float at12Shared;
	__shared__ bool mustContinue;


	if (thIdx == 0) {
		Ap1Shared = Ap1[i];
		CpShared = Cp[i];
		Ap12Shared = Ap12[i];
		PmaxShared = Pmax[i];
		PminShared = Pmin[i];
		nVoisinShared = nVoisin[i];
		at1Shared = at1;
		at2Shared = at2;
		at12Shared = at1 + at2;
		MuShared = MU[i];
		TMoyShared = Tmoy[i];
		PShared = P[i];
		mustContinue = false;
	}
	int k = 0;
	__syncthreads();
	const int CoresAgentLinLocal = CoresAgentLin[i];
	const int beginLocal = CoresAgentLinLocal + thIdx;
	const int endLocal = CoresAgentLinLocal + nVoisinShared;
	float res;
	for (int j = beginLocal; j < endLocal; j += step) {
		Bt1local[k] = Bt1[j];
		Ctlocal[k] = Ct[j];
		matUblocal[k] = matUb[j];
		matLblocal[k] = matLb[j];
		//Tlocalprelocal[k] = Tlocal_pre[j];
		Tlocallocal[k] = Tlocal_pre[j];
		k = k + 1;
	}

	__shared__ float shArr[blockSize];

	//Calcul des it�rations

	for (int iter = 0; iter < nStepL; iter++) {

		MULOCAL = MuShared; // tous lisent le m�me : broadcast !
		moy = TMoyShared;
		p = PShared;
		sum = 0;
		k = 0;
		for (int j = beginLocal; j < endLocal; j += step) {
			Tlocalprelocal[k] = Tlocallocal[k];
			m = Tlocallocal[k] - moy + p - MULOCAL;
			r = (Bt1local[k] * at1Shared + m * at2Shared - Ctlocal[k]) / (at12Shared);
			ub = matUblocal[k];
			lb = matLblocal[k];
			t = (ub - r) * (r > ub) + (lb - r) * (r < lb) + r;
			Tlocallocal[k] = t;
			sum += t;
			res = (t - Tlocalprelocal[k]);
			res = (double) res*res;
			if (res > eps) {
				mustContinue = true; // pas de race condition, car l'ordre n'importe pas,
				//mais est ce que cela ne va pas physiquement bloquer ?
			}
			k = k + 1;
		}

		shArr[thIdx] = sum;
		__syncthreads();
		if (blockSize >= 512) { if (thIdx < 256) { shArr[thIdx] += shArr[thIdx + 256]; } __syncthreads(); }
		if (blockSize >= 256) { if (thIdx < 128) { shArr[thIdx] += shArr[thIdx + 128]; } __syncthreads(); }
		if (blockSize >= 128) { if (thIdx < 64) { shArr[thIdx] += shArr[thIdx + 64]; } __syncthreads(); }
		if (thIdx < 32) {
			warpReduce<blockSize>(shArr, thIdx);
		}
		__syncthreads();

		if (thIdx == 0) {
			moy = shArr[0] / nVoisinShared;
			TMoyShared = moy;
			bp = moy + MuShared;
			p = (Ap1Shared * bp - CpShared) / Ap12Shared;
			p = (PmaxShared - p) * (p > PmaxShared) + (PminShared - p) * (p < PminShared) + p;
			PShared = p;
			res = p - moy;
			res = (double) res* res;
			if (res > eps) {
				mustContinue = true;
			}
			MuShared = MULOCAL + moy - p;
		}
		__syncthreads();
		if (!mustContinue) {
			break;
		}
		else {
			__syncthreads();
			if (thIdx == 0) {
				mustContinue = false;
			}
		}
	}
	//Ecriture des it�rations
	__syncthreads();
	k = 0;
	for (int j = beginLocal; j < endLocal; j += step) {
		Tlocal[j] = Tlocallocal[k];
		Tlocal_pre[j] = Tlocalprelocal[k];
		k = k + 1;
	}
	if (thIdx == 0) {
		Tmoy[blockIdx.x] = TMoyShared;// TMoyShared;
		P[blockIdx.x] = PShared;// PShared;
		MU[blockIdx.x] = MuShared;// MuShared;
	}

}











*/









