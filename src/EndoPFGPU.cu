#include "hip/hip_runtime.h"
#include "../head/EndoPFGPU.cuh"
 

// On prend la transpose de G !!! (ie G(n,i) = G[n*Nvar + i] )


EndoPFGPU::EndoPFGPU() : MethodP2PGPU()
{
#if DEBUG_CONSTRUCTOR
	std::cout << " EndoPFGPU Constructor" << std::endl;
#endif // DEBUG_CONSTRUCTOR
	_name = NAME;
	timePerBlock = MatrixCPU(1, 9, 0); // Fb0, Fb1 , Fb2, Fb3, Fb5, Fb6 Fb0'
// si les sous ensemble ne sont pas accessible, tout est dans le premier.
	occurencePerBlock = MatrixCPU(1, 9, 0); //nb de fois utilis� pendant la simu
}


EndoPFGPU::EndoPFGPU(float rho) : MethodP2PGPU()
{
#if DEBUG_CONSTRUCTOR
	std::cout << "default EndoPFGPU Constructor" << std::endl;
#endif // DEBUG_CONSTRUCTOR
	_name = NAME;
	_rho = rho;
	timePerBlock = MatrixCPU(1, 9, 0); // Fb0, Fb1 , Fb2, Fb3, Fb5, Fb6 Fb0'
// si les sous ensemble ne sont pas accessible, tout est dans le premier.
	occurencePerBlock = MatrixCPU(1, 9, 0); //nb de fois utilis� pendant la simu
}

EndoPFGPU::~EndoPFGPU()
{
	DELETEB(pf);
}
void EndoPFGPU::setParam(float rho)
{
	_rho = rho;
}

void EndoPFGPU::setTau(float tau)
{
	if (tau < 1) {
		throw std::invalid_argument("tau must be greater than 1");
	}
	_tau = tau;
}



void EndoPFGPU::solve(Simparam* result, const Simparam& sim, const StudyCase& cas)
{
#ifdef DEBUG_SOLVE
	cas.display();
	sim.display(1);
#endif // DEBUG_SOLVE
	
	tMarket = clock();
#ifdef INSTRUMENTATION
	std::chrono::high_resolution_clock::time_point t1;
	std::chrono::high_resolution_clock::time_point t2;
#endif // INSTRUMENTATION


	// FB 0
	if (_id == 0) {
#ifdef INSTRUMENTATION
		t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION
		init(sim, cas);
#ifdef INSTRUMENTATION
		hipDeviceSynchronize();
		t2 = std::chrono::high_resolution_clock::now();
		timePerBlock.set(0, 0, (float) std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
		occurencePerBlock.set(0, 0, 1);
#endif // INSTRUMENTATION
	}
	_rhog = sim.getRho();
	_at1 = _rhog;
	
	float resG = 2 * _epsG;
	float epsL2 = _epsL * _epsL;
	_iterGlobal = 0;
	//CHECK_LAST_CUDA_ERROR();
	//Pn.display(true);
	//std::cout << "*******" << std::endl;
	while ((_iterGlobal < _iterG) && (resG>_epsG) || (_iterGlobal <= _stepG)) {
		
#ifdef INSTRUMENTATION
		hipDeviceSynchronize();
		t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION
		updateLocalProbGPU(epsL2, _iterL);
		//CHECK_LAST_CUDA_ERROR();
#ifdef INSTRUMENTATION
		hipDeviceSynchronize();
		t2 = std::chrono::high_resolution_clock::now();
		timePerBlock.increment(0, 1, (float) std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
#endif // INSTRUMENTATION
		
		tradeLin.swap(&Tlocal); // echange juste les pointeurs	


		updateGlobalProbGPU();
		//CHECK_LAST_CUDA_ERROR();
		if (!(_iterGlobal % _stepG)) {
#ifdef INSTRUMENTATION
			hipDeviceSynchronize();
			t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION
			resG = updateResEndo(_iterGlobal / _stepG);
			//CHECK_LAST_CUDA_ERROR();
#ifdef INSTRUMENTATION
			hipDeviceSynchronize();
			t2 = std::chrono::high_resolution_clock::now();
			timePerBlock.increment(0, 6, (float) std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
#endif // INSTRUMENTATION
		}
		_iterGlobal++;
	}
	//std::cout << _iterGlobal << " " << resG << " " << resF.get(0, (_iterGlobal - 1) / _stepG) << " " << resF.get(1, (_iterGlobal - 1) / _stepG) << " " << resF.get(2, (_iterGlobal - 1) / _stepG) << std::endl;
#ifdef INSTRUMENTATION
	occurencePerBlock.increment(0, 1, _iterGlobal);
	occurencePerBlock.increment(0, 3, _iterGlobal);
	occurencePerBlock.increment(0, 4, _iterGlobal);
	occurencePerBlock.increment(0, 5, _iterGlobal);
	occurencePerBlock.increment(0, 6, _iterGlobal / _stepG);

	hipDeviceSynchronize();
	t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION
	


	// FB 5
	setResult(result, cas.isAC());

#ifdef INSTRUMENTATION
	t2 = std::chrono::high_resolution_clock::now();
	timePerBlock.increment(0, 7, (float) std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
	occurencePerBlock.increment(0, 7, 1);
	result->setTimeBloc(&timePerBlock, &occurencePerBlock);
#endif // INSTRUMENTATION
	
}

void EndoPFGPU::init(const Simparam& sim, const StudyCase& cas)
{
	DELETEB(pf);
	//std::cout << "init " << std::endl;
	isAC = true;
	if (!cas.isAC()) {
		throw std::invalid_argument("EndoPFGPU::init : Wrong studyCase must be AC");
	}
	
	initSize(cas);
	//_nLine = cas.getNLine(true);
	_nVarPF = _nLine + 2 * _nBus;
	initSimParam(sim);
	
	tempL1 = MatrixGPU(_nVarPF, 1, 0, 1);
	tempL1.preallocateReduction();
	
	isRadial = cas.isRadial();
	
	int nVoisinMax = nVoisin.max2();
	if (_blockSize * NMAXPEERPERTRHREAD < nVoisinMax) {
		std::cout << _blockSize << " " << NMAXPEERPERTRHREAD << " " << nVoisinMax << std::endl;
		throw std::invalid_argument("EndoPFGPU::init For this Method, an agent must not have more than 5120 peers");
	}

	_numBlocksBL = MYMAX(ceil((_nVarPF + _blockSize - 1) / _blockSize), 1);
	
	
	//std::cout << _numBlocksN << " " << _numBlocksM << " " << _numBlocksL << " " << _numBlocksBL << std::endl;
	//std::cout <<  _blockSize << std::endl;
	initCaseParam(sim, cas);
	if (initWithMarketClear) {
		ADMMMarketGPU market;
		Simparam res(sim);
		market.solve(&res, sim, cas);
		//res.display();
		LAMBDA = res.getLambda();
		trade = res.getTrade();
		Pn = MatrixGPU(res.getPn(), 1);
		Tmoy = Pn;

	}
	Pnpre = Pn;
	//Pnpre.display(true);
	
	//std::cout << "mise sous forme lineaire" << std::endl;
	initLinForm(cas);
	
	//std::cout << "donnees sur CPU pour le grid" << std::endl;
	delta1 = MatrixGPU(_nVarPF, 1, 0, 1);
	delta2 = MatrixGPU(_nVarPF, 1, 0, 1);
	Z1 = MatrixGPU(_nVarPF, 1, 0, 1);
	Z2 = MatrixGPU(_nVarPF, 1, 0, 1);
	Y = MatrixGPU(_nVarPF, 1, 0, 1);
	Ypre = MatrixGPU(_nVarPF, 1, 0, 1);
	dY = MatrixGPU(_nVarPF, 1, 0, 1);

	Ylimit = MatrixGPU(_nVarPF, 1, 0, 1); // angle, amplitude, flux
	YOffset = MatrixGPU(_nVarPF, 1, 0, 1); // angle, amplitude, flux
	G = MatrixGPU(_nAgent, _nVarPF, 0, 1);
	SensiBis = MatrixGPU(_nVarPF, 1, 0, 1);
	
	
	MatrixGPU LimitsUb(cas.getUpperBound(), 1); // angle, amplitude, flux
	MatrixGPU LimitsLb(cas.getLowerBound(), 1);
	
	initLimits << <_numBlocksBL, _blockSize >> > (Ylimit._matrixGPU, YOffset._matrixGPU, LimitsLb._matrixGPU, LimitsUb._matrixGPU, _nVarPF);
	//Ylimit.display(true);
	//YOffset.display(true);
	
	//CHECK_LAST_CUDA_ERROR();

	//std::cout << " PF " << std::endl;
	if (isRadial) {
		pf = new GPUPFdistPQ;
	}
	else {
		pf = new GPUPF;
	}
	//Ylimit.display();
	//YOffset.display();

	pf->init(cas, &Pn);

	
	//CHECK_LAST_CUDA_ERROR();
	//std::cout << "autres donnee sur CPU" << std::endl;
	
	initP2PMarket();
	//std::cout << "*******" << std::endl;
	dP = MatrixGPU(_nAgent, 1, 0, 1);
	Cp2 = MatrixGPU(_nAgent, 1, 0, 1);
	Cp1 = b;
	Cp1.multiplyT(&nVoisin);
	
	
	//std::cout << "update Global" << std::endl;
	updateGlobalProbGPU();
	//std::cout << "rho " << _rhog << " rhoL " << _rhol << " rho1 " << _rho1 << std::endl;
	//std::cout << "fin init temps : " << (float)(clock() - t) / CLOCKS_PER_SEC << std::endl;
	//std::cout << "fin init " << std::endl;
	CHECK_LAST_CUDA_ERROR();

}

void EndoPFGPU::updateGlobalProbGPU() {
	
	// FB 2a
#ifdef INSTRUMENTATION
	hipDeviceSynchronize();
	std::chrono::high_resolution_clock::time_point t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION
	//std::cout << "*************" << std::endl;
	
	Pn.swap(&Pnpre);
	updatePn();
	//std::cout << " Tmoy " << std::endl;
	
	//std::cout << " Pn " << std::endl;
	//Pn.display(true);
//	tradeLin.display(true);
	
	//std::cout << "update PF" << std::endl;
	pf->updatePQ(&Pn);
	
#ifdef INSTRUMENTATION
	hipDeviceSynchronize();
	std::chrono::high_resolution_clock::time_point t2 = std::chrono::high_resolution_clock::now();

	timePerBlock.increment(0, 3, (float) std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());

	// FB 3b
	hipDeviceSynchronize();
	t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION	
	 

	//std::cout << "solve PF" << std::endl;
	
	// FB 3
	pf->solve();
	 

	tempL1 = pf->getY();
	Ypre.swap(&Y);
	Y.subtract(&tempL1, &YOffset);
	
#ifdef INSTRUMENTATION
	hipDeviceSynchronize();
	t2 = std::chrono::high_resolution_clock::now();
	timePerBlock.increment(0, 4, (float) std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());

	// FB 3c
	hipDeviceSynchronize();
	t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION
	//std::cout << "update PF" << std::endl;
	//std::cout << "Y " << std::endl;
	//Y.display(true);
	
	float Ploss = - pf->getPloss() ;
	float Qloss = - pf->getQloss();	
	//std::cout << " Ploss " << Ploss << " Qloss " << Qloss << std::endl;
	
	Pmin.set(0, 0, Ploss / nVoisinCPU.get(0, 0), true);
	Pmax.set(0, 0, Ploss / nVoisinCPU.get(0, 0), true);
	
	Pmin.set(_nAgentTrue, 0, Qloss / nVoisinCPU.get(_nAgentTrue, 0), true);
	Pmax.set(_nAgentTrue, 0, Qloss / nVoisinCPU.get(_nAgentTrue, 0), true);
	
	if (Qloss > 0) {
		matUb.setBloc(_nTradeP, _nTradeP + _nAgentTrue - 1, 0, 1, Qloss);
	}
	else {
		matLb.setBloc(_nTradeP, _nTradeP + _nAgentTrue - 1, 0, 1, Qloss);
	}/**/
	
 
	


	updateZDeltaGPU << <_numBlocksBL, _blockSize >> > (Z1._matrixGPU, Z2._matrixGPU, Ylimit._matrixGPU, delta1._matrixGPU, delta2._matrixGPU, Y._matrixGPU, _nVarPF);

	updateSensi();
	
	updateCp2();
	Cp.add(&Cp1, &Cp2);
/*std::cout << " Z " << std::endl;
	Z1.display(true);
	Z2.display(true);

	std::cout << " Delta " << std::endl;
	delta1.display(true);
	delta2.display(true);
std::cout << " Sensi " << std::endl;
	SensiBis.display(true);
	G.display(true);	
	std::cout << " Cp " << std::endl;
	Cp2.display(true);
	//Cp.display(true);
*/
	////CHECK_LAST_CUDA_ERROR();

	

	// FB 3c
	updateLAMBDABt1GPU << <_numBlocksM, _blockSize >> > (Bt1._matrixGPU, LAMBDALin._matrixGPU, tradeLin._matrixGPU, _rhog, CoresLinTrans._matrixGPU, _nTrade);
	 
	
#ifdef INSTRUMENTATION
	t2 = std::chrono::high_resolution_clock::now();
	timePerBlock.increment(0, 5, (float) std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
#endif // INSTRUMENTATION
	
}

void EndoPFGPU::updateLocalProbGPU(float epsL, int nIterL) {
	// FB 1a
	int numBlocks = _nAgent;
	/*std::cout << "probl�me local" << std::endl;
	std::cout << _at1 << " " << _at2 << std::endl;
	Bt1.display(true);
	Ct.display(true);
	matLb.display(true);
	matUb.display(true);
	Ap1.display(true);
	Ap12.display(true);
	Bp1.display(true);
	Cp.display(true);
	Pmin.display(true);
	Pmax.display(true);*/


	switch (_blockSize) {
	case 512:
		updateTradePGPUSharedResidual<512> << <numBlocks, _blockSize >> > (Tlocal._matrixGPU, Tlocal_pre._matrixGPU, Tmoy._matrixGPU, P._matrixGPU, MU._matrixGPU, nVoisin._matrixGPU, _at1, _at2, Bt1._matrixGPU, Ct._matrixGPU,
			matLb._matrixGPU, matUb._matrixGPU, Ap1._matrixGPU, Ap12._matrixGPU, Cp._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, CoresAgentLin._matrixGPU, epsL, nIterL);
		break;
	case 256:
		updateTradePGPUSharedResidual<256> << <numBlocks, _blockSize >> > (Tlocal._matrixGPU, Tlocal_pre._matrixGPU, Tmoy._matrixGPU, P._matrixGPU, MU._matrixGPU, nVoisin._matrixGPU, _at1, _at2, Bt1._matrixGPU, Ct._matrixGPU,
			matLb._matrixGPU, matUb._matrixGPU, Ap1._matrixGPU, Ap12._matrixGPU, Cp._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, CoresAgentLin._matrixGPU, epsL, nIterL);
		break;
	case 128:
		updateTradePGPUSharedResidual<128> << <numBlocks, _blockSize >> > (Tlocal._matrixGPU, Tlocal_pre._matrixGPU, Tmoy._matrixGPU, P._matrixGPU, MU._matrixGPU, nVoisin._matrixGPU, _at1, _at2, Bt1._matrixGPU, Ct._matrixGPU,
			matLb._matrixGPU, matUb._matrixGPU, Ap1._matrixGPU, Ap12._matrixGPU, Cp._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, CoresAgentLin._matrixGPU, epsL, nIterL);
		break;
	case 64:
		updateTradePGPUSharedResidual< 64> << <numBlocks, _blockSize >> > (Tlocal._matrixGPU, Tlocal_pre._matrixGPU, Tmoy._matrixGPU, P._matrixGPU, MU._matrixGPU, nVoisin._matrixGPU, _at1, _at2, Bt1._matrixGPU, Ct._matrixGPU,
			matLb._matrixGPU, matUb._matrixGPU, Ap1._matrixGPU, Ap12._matrixGPU, Cp._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, CoresAgentLin._matrixGPU, epsL, nIterL);
		break;
	case 32:
		updateTradePGPUSharedResidual< 32> << <numBlocks, _blockSize >> > (Tlocal._matrixGPU, Tlocal_pre._matrixGPU, Tmoy._matrixGPU, P._matrixGPU, MU._matrixGPU, nVoisin._matrixGPU, _at1, _at2, Bt1._matrixGPU, Ct._matrixGPU,
			matLb._matrixGPU, matUb._matrixGPU, Ap1._matrixGPU, Ap12._matrixGPU, Cp._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, CoresAgentLin._matrixGPU, epsL, nIterL);
		break;
	case 16:
		updateTradePGPUSharedResidual< 16> << <numBlocks, _blockSize >> > (Tlocal._matrixGPU, Tlocal_pre._matrixGPU, Tmoy._matrixGPU, P._matrixGPU, MU._matrixGPU, nVoisin._matrixGPU, _at1, _at2, Bt1._matrixGPU, Ct._matrixGPU,
			matLb._matrixGPU, matUb._matrixGPU, Ap1._matrixGPU, Ap12._matrixGPU, Cp._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, CoresAgentLin._matrixGPU, epsL, nIterL);
		break;
	case  8:
		updateTradePGPUSharedResidual<  8> << <numBlocks, _blockSize >> > (Tlocal._matrixGPU, Tlocal_pre._matrixGPU, Tmoy._matrixGPU, P._matrixGPU, MU._matrixGPU, nVoisin._matrixGPU, _at1, _at2, Bt1._matrixGPU, Ct._matrixGPU,
			matLb._matrixGPU, matUb._matrixGPU, Ap1._matrixGPU, Ap12._matrixGPU, Cp._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, CoresAgentLin._matrixGPU, epsL, nIterL);
		break;
	case  4:
		updateTradePGPUSharedResidual<  4> << <numBlocks, _blockSize >> > (Tlocal._matrixGPU, Tlocal_pre._matrixGPU, Tmoy._matrixGPU, P._matrixGPU, MU._matrixGPU, nVoisin._matrixGPU, _at1, _at2, Bt1._matrixGPU, Ct._matrixGPU,
			matLb._matrixGPU, matUb._matrixGPU, Ap1._matrixGPU, Ap12._matrixGPU, Cp._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, CoresAgentLin._matrixGPU, epsL, nIterL);
		break;
	case  2:
		updateTradePGPUSharedResidual<  2> << <numBlocks, _blockSize >> > (Tlocal._matrixGPU, Tlocal_pre._matrixGPU, Tmoy._matrixGPU, P._matrixGPU, MU._matrixGPU, nVoisin._matrixGPU, _at1, _at2, Bt1._matrixGPU, Ct._matrixGPU,
			matLb._matrixGPU, matUb._matrixGPU, Ap1._matrixGPU, Ap12._matrixGPU, Cp._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, CoresAgentLin._matrixGPU, epsL, nIterL);
		break;
	case  1:
		updateTradePGPUSharedResidual<  1> << <numBlocks, _blockSize >> > (Tlocal._matrixGPU, Tlocal_pre._matrixGPU, Tmoy._matrixGPU, P._matrixGPU, MU._matrixGPU, nVoisin._matrixGPU, _at1, _at2, Bt1._matrixGPU, Ct._matrixGPU,
			matLb._matrixGPU, matUb._matrixGPU, Ap1._matrixGPU, Ap12._matrixGPU, Cp._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, CoresAgentLin._matrixGPU, epsL, nIterL);
		break;
	}

	

}


void EndoPFGPU::updateCp2()
{
	int numBlocks = _nAgent;
	switch (_blockSize) {
	case 512:
		updateCp2GPU<512> << <numBlocks, _blockSize >> > (Cp2._matrixGPU, SensiBis._matrixGPU, G._matrixGPU, nVoisin._matrixGPU, _rho1, _nVarPF);
		break;
	case 256:
		updateCp2GPU<256> << <numBlocks, _blockSize >> > (Cp2._matrixGPU, SensiBis._matrixGPU, G._matrixGPU, nVoisin._matrixGPU, _rho1, _nVarPF);
		break;
	case 128:
		updateCp2GPU<128> << <numBlocks, _blockSize >> > (Cp2._matrixGPU, SensiBis._matrixGPU, G._matrixGPU, nVoisin._matrixGPU, _rho1, _nVarPF);
		break;
	case 64:
		updateCp2GPU< 64> << <numBlocks, _blockSize >> > (Cp2._matrixGPU, SensiBis._matrixGPU, G._matrixGPU, nVoisin._matrixGPU, _rho1, _nVarPF);
		break;
	case 32:
		updateCp2GPU< 32> << <numBlocks, _blockSize >> > (Cp2._matrixGPU, SensiBis._matrixGPU, G._matrixGPU, nVoisin._matrixGPU, _rho1, _nVarPF);
		break;
	case 16:
		updateCp2GPU< 16> << <numBlocks, _blockSize >> > (Cp2._matrixGPU, SensiBis._matrixGPU, G._matrixGPU, nVoisin._matrixGPU, _rho1, _nVarPF);
		break;
	case  8:
		updateCp2GPU<  8> << <numBlocks, _blockSize >> > (Cp2._matrixGPU, SensiBis._matrixGPU, G._matrixGPU, nVoisin._matrixGPU, _rho1, _nVarPF);
		break;
	case  4:
		updateCp2GPU<  4> << <numBlocks, _blockSize >> > (Cp2._matrixGPU, SensiBis._matrixGPU, G._matrixGPU, nVoisin._matrixGPU, _rho1, _nVarPF);
		break;
	case  2:
		updateCp2GPU<  2> << <numBlocks, _blockSize >> > (Cp2._matrixGPU, SensiBis._matrixGPU, G._matrixGPU, nVoisin._matrixGPU, _rho1, _nVarPF);
		break;
	case  1:
		updateCp2GPU<  1> << <numBlocks, _blockSize >> > (Cp2._matrixGPU, SensiBis._matrixGPU, G._matrixGPU, nVoisin._matrixGPU, _rho1, _nVarPF);
		break;
	}



}



void EndoPFGPU::updateSensi()
{

	updateSensiBis<<<_numBlocksBL,_blockSize>>> (SensiBis._matrixGPU, Y._matrixGPU, Z1._matrixGPU, Z2._matrixGPU, delta1._matrixGPU, delta2._matrixGPU, _nVarPF);
	dY.subtract(&Y, &Ypre);
	dP.subtract(&Pn, &Pnpre);
	//dY.display(true);
	//dP.display(true);
	updateSensiGPU << <_nAgent, _blockSize >> > (G._matrixGPU, dY._matrixGPU, dP._matrixGPU, _nVarPF);
	//G.display();
}




float EndoPFGPU::updateResEndo(int iter)
{
	
	float resS = Tlocal.max2(&tradeLin);

	updateDiffGPU << <_numBlocksM, _blockSize >> > (tempNN._matrixGPU, tradeLin._matrixGPU, CoresLinTrans._matrixGPU, _nTrade);
	float resR = tempNN.max2();
	// Residus reseau
	
	tempL1.subtract(&Ylimit, &Y);
	tempL1.projectNeg();

	float resXf = _ratioEps * tempL1.max2();
	resF.set(0, iter, resR);
	resF.set(1, iter, resS);
	resF.set(2, iter, resXf);

	return MYMAX(MYMAX(resXf, resS), resR);
}

void EndoPFGPU::display() {

	a.transferCPU();
	b.transferCPU();
	Pn.transferCPU();
	Pmin.transferCPU();
	Pmax.transferCPU();

	if (_iterGlobal == 0) {
		std::cout << "algorithm not launch" << std::endl;
	}
	else if (_iterGlobal < _iterG) {
		std::cout << "method " << _name << " converged in " << _iterGlobal << " iterations." << std::endl;
		std::cout << "Converged in " << (float) tMarket / CLOCKS_PER_SEC << " seconds" << std::endl;

	}
	else {
		std::cout << "method " << _name << " not converged in " << _iterGlobal << " iterations." << std::endl;
		std::cout << "time taken " << (float) tMarket / CLOCKS_PER_SEC << " seconds" << std::endl;
	}
	std::cout << "The power error of this state is (constraint) " << resF.get(0, _iterGlobal / _stepG) << " and convergence " << resF.get(1, _iterGlobal / _stepG) << std::endl;
	std::cout << "===============================================================|" << std::endl;
	std::cout << "      System Summary                                           |" << std::endl;
	std::cout << "===============================================================|" << std::endl;
	std::cout << "Agent            " << _nAgentTrue << std::endl;



	std::cout << std::endl << std::endl;
	std::cout << std::endl << std::endl;
	std::cout << "========================================================================================================|" << std::endl;
	std::cout << "      Agent Data                                                                                        |" << std::endl;
	std::cout << "========================================================================================================|" << std::endl;
	std::cout << " Agent  |  Cost    |  Cost    |          Power Injection           |           Power Injection          |" << std::endl;
	std::cout << "  #     |   a (pu) |   b (pu) |  P (pu)  | Pmin (pu)  | Pmax (pu)  |  Q (pu)   | Qmin (pu)  | Qmax (pu) |" << std::endl;
	std::cout << "--------|----------|----------|----------|------------|------------|-----------|------------|-----------|" << std::endl;

	for (int n = 0; n < _nAgentTrue; n++) {

		std::cout << std::setw(8) << n << "|" << std::setw(9) << a.get(n, 0) << " |" << std::setw(10)
			<< b.get(n, 0) << "|" << std::setw(10) << Pn.get(n, 0) << "|" << std::setw(12)
			<< Pmin.get(n, 0) * nVoisinCPU.get(n, 0) << "|" << std::setw(12) << Pmax.get(n, 0) * nVoisinCPU.get(n, 0)
			<< "|" << std::setw(11) << Pn.get(n + _nAgentTrue, 0) << "|" << std::setw(12) << Pmin.get(n + _nAgentTrue, 0) * nVoisinCPU.get(n + _nAgentTrue, 0)
			<< "|" << std::setw(11) << Pmax.get(n + _nAgentTrue, 0) * nVoisinCPU.get(n + _nAgentTrue, 0) << "|" << std::endl;
	}


	std::cout << "========================================================================================================|" << std::endl;
	std::cout << "                      END PRINT                                                                         |" << std::endl;
	std::cout << "========================================================================================================|" << std::endl;


	a.transferGPU();
	b.transferGPU();
	Pn.transferGPU();
	Pmin.transferGPU();
	Pmax.transferGPU();
}





__global__ void initLimits(float* Ylimit, float* Yoffset, float* limitsLb, float* limitsUb, int nVarPF) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int step = blockDim.x * gridDim.x;

	for (int i = index; i < nVarPF; i += step) {
		float ub = limitsUb[i];
		float lb = limitsLb[i];
		float mid = (ub + lb) / 2;
		float lim = ub - mid;
		Ylimit[i] = lim;
		Yoffset[i] = mid;
	}
}



/*
	Z1.add(&Ylimit, &delta1);
	Z1.subtract(&Y);
	Z1.projectPos();

	Z2.add(&Ylimit, &delta2);
	Z2.add(&Y);
	Z2.projectPos();
*/

__global__ void updateZGPU(float* Z1, float* Z2, float* Ylimit, float* delta1, float* delta2, float* Y, int nVarPF) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int step = blockDim.x * gridDim.x;

	for (int i = index; i < nVarPF; i += step) {

		float z1 = Ylimit[i] + delta1[i] - Y[i];
		float z2 = Ylimit[i] + delta2[i] + Y[i];
		Z1[i] = (z1 > 0) * z1;
		Z2[i] = (z2 > 0) * z2;

	}
}


/*
	delta1.add(&Ylimit);
	delta1.subtract(&Z1);
	delta1.subtract(&Y);

	delta2.add(&Ylimit);
	delta2.subtract(&Z2);
	delta2.add(&Y);
*/
__global__ void updateDeltaGPU(float* delta1, float* delta2, float* Z1, float* Z2, float* Ylimit, float* Y, int nVarPF) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int step = blockDim.x * gridDim.x;

	for (int i = index; i < nVarPF; i += step) {
		float d1 = Ylimit[i] + delta1[i] - Y[i];
		float d2 = Ylimit[i] + delta2[i] + Y[i];

		delta1[i] = (d1 < 0) * d1;//delta1[i] + Ylimit[i] - Z1[i] - Y[i];
		delta2[i] = (d2 < 0) * d2;//delta2[i] + Ylimit[i] - Z2[i] + Y[i];
	}
}


__global__ void updateZDeltaGPU(float* Z1, float* Z2, float* Ylimit, float* delta1, float* delta2, float* Y, int nVarPF) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int step = blockDim.x * gridDim.x;

	for (int i = index; i < nVarPF; i += step) {

		float z1 = Ylimit[i] + delta1[i] - Y[i];
		float z2 = Ylimit[i] + delta2[i] + Y[i];
		Z1[i] = (z1 > 0) * z1;
		Z2[i] = (z2 > 0) * z2;
		delta1[i] = (z1 < 0) * z1;
		delta2[i] = (z2 < 0) * z2;
	}

}



/*
for (int n = 0; n < _nAgent; n++) {
		float sum = 0;
		for (int i = 0; i < _nVarPF; i++) {
			sum += SensiBis.get(i, 0) * G.get(i, n);
		}
		Cp2.set(n, 0, sum * _rho1 * nVoisin.get(n, 0));
	}

*/

template <unsigned int blockSize>
__global__ void updateCp2GPU(float* Cp2, float* SensiBis, float* G, float* nVoisin, float rho1, int nVarPF) {
	int index = threadIdx.x;
	int step = blockDim.x;
	int agent = blockIdx.x; // un bloc par agent
	__shared__ float shArr[blockSize];

	float sum = 0;
	for (int i = index; i < nVarPF; i += step) {
		sum += SensiBis[i] * G[agent * nVarPF + i];
	}
	shArr[index] = sum;
	__syncthreads();

	if (blockSize >= 512) { if (index < 256) { shArr[index] += shArr[index + 256]; } __syncthreads(); }
	if (blockSize >= 256) { if (index < 128) { shArr[index] += shArr[index + 128]; } __syncthreads(); }
	if (blockSize >= 128) { if (index < 64) { shArr[index] += shArr[index + 64]; } __syncthreads(); }
	if (index < 32) {
		warpReduce<blockSize>(shArr, index);
	}
	if (index == 0) {
		Cp2[agent] = shArr[0] * rho1 * nVoisin[agent];
	}

}


/*
	SensiBis.set(&Y);
	SensiBis.multiply(2);
	SensiBis.add(&Z1);
	SensiBis.subtract(&Z2);
	SensiBis.add(&delta2);
	SensiBis.subtract(&delta1);
*/

__global__ void updateSensiBis(float* sensiBis, float* Y, float* Z1, float* Z2, float* delta1, float* delta2, int nVarPF) {

	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int step = blockDim.x * gridDim.x;

	for (int i = index; i < nVarPF; i += step) {
		sensiBis[i] = 2 * Y[i] + Z1[i] - Z2[i] + delta2[i] - delta1[i];
	}

}



/*

	for (int i = 0; i < _nVarPF; i++) {
		for (int n = 1; n < _nAgent; n++) {
			if (abs(dP.get(n,0)) > 0.01) {
				G.set(i, n, dY.get(i, 0) / dP.get(n, 0));
			}
		}
	}

*/

__global__ void updateSensiGPU(float* G, float* dY, float* dP, int nVar) {
	int index = threadIdx.x;
	int step = blockDim.x;
	int agent = blockIdx.x; // un bloc par agent
	float dp = dP[agent];

	if (dp > 0.01 || dp < -0.01) {
		for (int i = index; i < nVar; i += step) {
			G[agent * nVar + i] = dY[i] / dp;
		}
	}



}







/*
template <unsigned int blockSize>
__global__ void updateTradePGPUSharedResidual(float* Tlocal, float* Tlocal_pre, float* Tmoy, float* P, float* MU, float* nVoisin, float at1, float at2, float* Bt1, float* Ct,
	float* matLb, float* matUb, float* Ap1, float* Ap12, float* Cp, float* Pmin, float* Pmax, float* CoresAgentLin, float eps, int nStepL) {

	//Definition de toutes les variables locales
	int i = blockIdx.x; // c'est aussi l'identifiant de l'agent !
	unsigned int thIdx = threadIdx.x;
	const int step = blockSize;
	// ne change pas


	float Bt1local[NMAXPEERPERTRHREAD];
	float Ctlocal[NMAXPEERPERTRHREAD];
	float matUblocal[NMAXPEERPERTRHREAD];
	float matLblocal[NMAXPEERPERTRHREAD];

	float Tlocallocal[NMAXPEERPERTRHREAD]; // change
	float Tlocalprelocal[NMAXPEERPERTRHREAD]; // change
	float sum;
	float bp, MULOCAL, moy, p;
	float m, r, ub, lb, t;
	// le changement doit �tre partag� par tous les threads du bloc

	__shared__ float MuShared;
	__shared__ float TMoyShared;
	__shared__ float PShared;


	// constant et commun � tous les thread d'un bloc
	__shared__ float Ap1Shared;
	__shared__ float CpShared;
	__shared__ float Ap12Shared;
	__shared__ float PmaxShared;
	__shared__ float PminShared;
	__shared__ float nVoisinShared;
	__shared__ float at1Shared;
	__shared__ float at2Shared;
	__shared__ float at12Shared;
	__shared__ bool mustContinue;


	if (thIdx == 0) {
		Ap1Shared = Ap1[i];
		CpShared = Cp[i];
		Ap12Shared = Ap12[i];
		PmaxShared = Pmax[i];
		PminShared = Pmin[i];
		nVoisinShared = nVoisin[i];
		at1Shared = at1;
		at2Shared = at2;
		at12Shared = at1 + at2;
		MuShared = MU[i];
		TMoyShared = Tmoy[i];
		PShared = P[i];
		mustContinue = false;
	}
	int k = 0;
	__syncthreads();
	const int CoresAgentLinLocal = CoresAgentLin[i];
	const int beginLocal = CoresAgentLinLocal + thIdx;
	const int endLocal = CoresAgentLinLocal + nVoisinShared;
	float res;
	for (int j = beginLocal; j < endLocal; j += step) {
		Bt1local[k] = Bt1[j];
		Ctlocal[k] = Ct[j];
		matUblocal[k] = matUb[j];
		matLblocal[k] = matLb[j];
		//Tlocalprelocal[k] = Tlocal_pre[j];
		Tlocallocal[k] = Tlocal_pre[j];
		k = k + 1;
	}

	__shared__ float shArr[blockSize];

	//Calcul des it�rations

	for (int iter = 0; iter < nStepL; iter++) {

		MULOCAL = MuShared; // tous lisent le m�me : broadcast !
		moy = TMoyShared;
		p = PShared;
		sum = 0;
		k = 0;
		for (int j = beginLocal; j < endLocal; j += step) {
			Tlocalprelocal[k] = Tlocallocal[k];
			m = Tlocallocal[k] - moy + p - MULOCAL;
			r = (Bt1local[k] * at1Shared + m * at2Shared - Ctlocal[k]) / (at12Shared);
			ub = matUblocal[k];
			lb = matLblocal[k];
			t = (ub - r) * (r > ub) + (lb - r) * (r < lb) + r;
			Tlocallocal[k] = t;
			sum += t;
			res = (t - Tlocalprelocal[k]);
			res = (double) res*res;
			if (res > eps) {
				mustContinue = true; // pas de race condition, car l'ordre n'importe pas,
				//mais est ce que cela ne va pas physiquement bloquer ?
			}
			k = k + 1;
		}

		shArr[thIdx] = sum;
		__syncthreads();
		if (blockSize >= 512) { if (thIdx < 256) { shArr[thIdx] += shArr[thIdx + 256]; } __syncthreads(); }
		if (blockSize >= 256) { if (thIdx < 128) { shArr[thIdx] += shArr[thIdx + 128]; } __syncthreads(); }
		if (blockSize >= 128) { if (thIdx < 64) { shArr[thIdx] += shArr[thIdx + 64]; } __syncthreads(); }
		if (thIdx < 32) {
			warpReduce<blockSize>(shArr, thIdx);
		}
		__syncthreads();

		if (thIdx == 0) {
			moy = shArr[0] / nVoisinShared;
			TMoyShared = moy;
			bp = moy + MuShared;
			p = (Ap1Shared * bp - CpShared) / Ap12Shared;
			p = (PmaxShared - p) * (p > PmaxShared) + (PminShared - p) * (p < PminShared) + p;
			PShared = p;
			res = p - moy;
			res = (double) res* res;
			if (res > eps) {
				mustContinue = true;
			}
			MuShared = MULOCAL + moy - p;
		}
		__syncthreads();
		if (!mustContinue) {
			break;
		}
		else {
			__syncthreads();
			if (thIdx == 0) {
				mustContinue = false;
			}
		}
	}
	//Ecriture des it�rations
	__syncthreads();
	k = 0;
	for (int j = beginLocal; j < endLocal; j += step) {
		Tlocal[j] = Tlocallocal[k];
		Tlocal_pre[j] = Tlocalprelocal[k];
		k = k + 1;
	}
	if (thIdx == 0) {
		Tmoy[blockIdx.x] = TMoyShared;// TMoyShared;
		P[blockIdx.x] = PShared;// PShared;
		MU[blockIdx.x] = MuShared;// MuShared;
	}

}











*/









