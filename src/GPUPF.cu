#include "hip/hip_runtime.h"
#include "../head/GPUPF.cuh"


GPUPF::GPUPF(){}
GPUPF::~GPUPF(){
   

}
void GPUPF::init(const StudyCase& cas, MatrixGPU* PQ)
{
#ifdef INSTRUMENTATION
    t1 = std::chrono::high_resolution_clock::now();
    timePerBlock = MatrixCPU(1, 9); // Fb0 : init, Fb1ab : Flu, Fb2abc: Tension , FB3 : puissance, Fb4 erreur, Fb0 mise � jour

    occurencePerBlock = MatrixCPU(1, 9);; //nb de fois utilis� pendant la simu
#endif // INSTRUMENTATION
    
   // std::cout << "init PF NR GPU simple" << std::endl;
    std::ios_base::openmode mode = std::fstream::in | std::fstream::out | std::fstream::app;
    Nagent = cas.getNagent();
    Nbus = cas.getNBus();
    B2 = 2 * Nbus;
    N2 = 2 * Nagent;
    Nline = cas.getNLine(true); // ne doit pas �tre r�duit ici !!!
    BL2 = Nbus + 2 * Nline;
    Nconstraint = B2 + Nline;
    iterM = 20;
    iter = 0;
    V0 = cas.getV0();
    theta0 = cas.gettheta0();
    I = MatrixGPU(cas.getCoresBusAgentLin(), 1);
    status = 0;

    CoresAgentBus = MatrixGPU(cas.getCoresAgentBusLin(), 1);
    CoresAgentBusBegin = MatrixGPU(cas.getCoresAgentBusLinBegin(), 1);
    NagentByBus = MatrixGPU(cas.getNagentByBus(), 1);
    removeLossAgent << <1, 1 >> > (NagentByBus._matrixGPU, CoresAgentBusBegin._matrixGPU);
    //I.display(true);
    _name = "Newton";
    numBlock = Nbus;
    _useDouble = false;


    Bgrid = cas.getLineSuceptance();
    Ggrid = cas.getLineReactance();



    Y = MatrixGPU(2 * Nbus + Nline, 1, 0, 1);
    CoresLineBus = MatrixGPU(cas.getCoresLineBus(true));
    _CoresVoiLin = MatrixGPU(cas.getCoresVoiLin(), 1);
    _CoresBusLin = MatrixGPU(cas.getCoresBusLin(), 1);
    _nLines = MatrixGPU(cas.getNLines(), 1);
    CoresLineBusGPU = MatrixGPU(2, Nline);

    for (int lold = 0; lold < Nline; lold++) {
        int busTo = CoresLineBus.get(lold, 1);
        int busFrom = CoresLineBus.get(lold, 0);
        CoresLineBusGPU.set(0, lold, busFrom);
        CoresLineBusGPU.set(1, lold, busTo);
    }
    CoresLineBusGPU.transferGPU();
    _Blin2 = MatrixGPU(cas.getBlin2(), 1);
    _Glin2 = MatrixGPU(cas.getGlin2(), 1);
    Phi = MatrixGPU(Nline, 1, 0, 1);
    //_nLines.display(true);


    
    _Blin = MatrixGPU(cas.getBlin(), 1);
    _Glin = MatrixGPU(cas.getGlin(), 1);

    W = MatrixGPU(B2, 1, 0, 1);
    _Pintermediate = MatrixGPU(BL2, 1, 0, 1);
    _Qintermediate = MatrixGPU(BL2, 1, 0, 1);
    dW = MatrixGPU(B2, 1, 0, 1);
    dW.preallocateReduction();
    E = MatrixGPU(B2, 1, 0, 1);
    dE = MatrixGPU(B2, 1, 0, 1);
    Jac = MatrixGPU(B2, B2);
    Jac.set(0, 0, 1);
    Jac.set(Nbus, Nbus, 1);
    Jac.transferGPU();
    JacInv = MatrixGPU(B2, B2, 0, 1);

    initE << <numBlock, _blockSize >> > (E._matrixGPU, theta0, V0, Nbus);
    //E.display(true);
    /*std::cout << " Bgrid : " << std::endl;
    Bgrid.display();
    std::cout << " Ggrid : " << std::endl;
    Ggrid.display();*/
    W0 = MatrixGPU(B2, 1, 0, 1);

    calculW0Bis(PQ);

    /*std::cout << " PQ : " << std::endl;
    PQ->display(true);
    std::cout << " W0 : " << std::endl;
    W0.display(true);

    CoresAgentBus.display(true);
    CoresAgentBusBegin.display(true);
    NagentByBus.display(true);
    std::cout << "N: " << Nagent << " B= " << Nbus << std::endl;*/


    //std::cout << " W0 : " << std::endl;
    //W0.display(true);
    calcW();

    dW.subtract(&W0, &W);
    A = MatrixGPU(B2, B2, 0, 1);
    P = MatrixGPU(B2 + 1, 1, 0, 1);

    
#ifdef INSTRUMENTATION
    hipDeviceSynchronize();
    t2 = std::chrono::high_resolution_clock::now();
    timePerBlock.increment(0, 0, std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
    occurencePerBlock.increment(0, 0, 1);
#endif // INSTRUMENTATION

    //std::cout << " fin init" << std::endl;

}
void GPUPF::init(const StudyCase& cas, MatrixGPU* PQ, MatrixGPUD* PQD, bool useDouble)
{
#ifdef INSTRUMENTATION
    t1 = std::chrono::high_resolution_clock::now();
    timePerBlock = MatrixCPU(1, 9); // Fb0 : init, Fb1ab : Flu, Fb2abc: Tension , FB3 : puissance, Fb4 erreur, Fb0 mise � jour

    occurencePerBlock = MatrixCPU(1, 9);; //nb de fois utilis� pendant la simu
#endif // INSTRUMENTATION
   // std::cout << "init PF NR GPU" << std::endl;
    std::ios_base::openmode mode = std::fstream::in | std::fstream::out | std::fstream::app;
    Nagent = cas.getNagent();
    Nbus = cas.getNBus();
    B2 = 2 * Nbus;
    N2 = 2 * Nagent;
    Nline = cas.getNLine(true); // ne doit pas �tre r�duit ici !!!
    BL2 = Nbus + 2 * Nline;
    Nconstraint = B2 + Nline;
    iterM = 20;
    iter = 0;
    V0 = cas.getV0();
    theta0 = cas.gettheta0();
    I = MatrixGPU(cas.getCoresBusAgentLin(),1);
    status = 0;

    CoresAgentBus = MatrixGPU(cas.getCoresAgentBusLin(), 1);
    CoresAgentBusBegin = MatrixGPU(cas.getCoresAgentBusLinBegin(), 1);
    NagentByBus = MatrixGPU(cas.getNagentByBus(), 1);
    //I.display(true);
    _name = "Newton";
    numBlock = Nbus;
    _useDouble = useDouble;
   

    Bgrid = cas.getLineSuceptance();
    Ggrid = cas.getLineReactance();

    
     
    Y = MatrixGPU(2 * Nbus + Nline, 1, 0, 1);
    CoresLineBus = MatrixGPU(cas.getCoresLineBus(true));
    _CoresVoiLin = MatrixGPU(cas.getCoresVoiLin(), 1);
    _CoresBusLin = MatrixGPU(cas.getCoresBusLin(), 1);
    _nLines = MatrixGPU(cas.getNLines(), 1);
    CoresLineBusGPU = MatrixGPU(2, Nline);
    
    for (int lold = 0; lold < Nline; lold++) {
        int busTo = CoresLineBus.get(lold, 1);
        int busFrom = CoresLineBus.get(lold, 0);
        CoresLineBusGPU.set(0, lold, busFrom);
        CoresLineBusGPU.set(1, lold, busTo);
    }
    CoresLineBusGPU.transferGPU();
    _Blin2 = MatrixGPU(cas.getBlin2(), 1);
    _Glin2 = MatrixGPU(cas.getGlin2(), 1);
    //_nLines.display(true);


    if (_useDouble) {
        
        _BlinD = MatrixGPUD(cas.getBlinD(), 1);
        _GlinD = MatrixGPUD(cas.getGlinD(), 1);

        WD = MatrixGPUD(B2, 1, 0, 1);
        _PintermediateD = MatrixGPUD(BL2, 1, 0, 1);
        _QintermediateD = MatrixGPUD(BL2, 1, 0, 1);
      

        dWD = MatrixGPUD(B2, 1, 0, 1);
        dWD.preallocateReduction();
        ED = MatrixGPUD(B2, 1, 0, 1);
        initED <<<numBlock,_blockSize>>>(ED._matrixGPU, theta0, V0, Nbus);
        
        //ED.display(true);

        dED = MatrixGPUD(B2, 1, 0, 1);
        JacD = MatrixGPUD(B2, B2);
        if (JacD.getPos()) {
            JacD.transferCPU();
        }
        JacD.set(0, 0, 1);
        JacD.set(Nbus, Nbus, 1);
        JacD.transferGPU();

        JacInvD = MatrixGPUD(B2, B2, 0, 1);
        W0D = MatrixGPUD(B2, 1, 0, 1);

        /*std::cout << " PQ : " << std::endl;
        PQD->display(true);
        std::cout << " W0 : " << std::endl;
        W0D.display(true);
       
        CoresAgentBus.display(true);
        CoresAgentBusBegin.display(true);
        NagentByBus.display(true);

        std::cout << "N: " << Nagent<< " B= "<< Nbus << std::endl;*/
        calculW0DBis(PQD);
        /*ED.display(true);
        _GlinD.display(true);
        _BlinD.display(true);
        std::cout << "------------" << std::endl;
        _CoresVoiLin.display(true);
        _CoresBusLin.display(true);
        _nLines.display(true);
        std::cout << "------------" << std::endl;

        _PintermediateD.display(true);
        _QintermediateD.display(true);*/
        //std::cout << "*******" << std::endl;



        //std::cout << " W0 : " << std::endl;
        //W0D.display(true);

        calcW();

        dWD.subtract(&W0D, &WD);
        AD = MatrixGPUD(B2, B2, 0, 1);
        PD = MatrixGPUD(B2 + 1, 1, 0, 1);
    
    }
    else {
        
        _Blin = MatrixGPU(cas.getBlin(), 1);
        _Glin = MatrixGPU(cas.getGlin(), 1);
           
        W = MatrixGPU(B2, 1, 0, 1);
        _Pintermediate = MatrixGPU(BL2, 1, 0, 1);
        _Qintermediate = MatrixGPU(BL2, 1, 0, 1);
        dW = MatrixGPU(B2, 1, 0, 1);
        dW.preallocateReduction();
        E = MatrixGPU(B2, 1, 0, 1);
        dE = MatrixGPU(B2, 1, 0, 1);
        Jac = MatrixGPU(B2, B2);
        Jac.set(0, 0, 1);
        Jac.set(Nbus, Nbus, 1);
        Jac.transferGPU();
        JacInv = MatrixGPU(B2, B2, 0, 1);

        initE << <numBlock, _blockSize >> > (E._matrixGPU, theta0, V0, Nbus);
        //E.display(true);
        /*std::cout << " Bgrid : " << std::endl;
        Bgrid.display();
        std::cout << " Ggrid : " << std::endl;
        Ggrid.display();*/
        W0 = MatrixGPU(B2, 1, 0, 1);

        calculW0Bis(PQ); 
       
        /*std::cout << " PQ : " << std::endl;
        PQ->display(true);
        std::cout << " W0 : " << std::endl;
        W0.display(true);

        CoresAgentBus.display(true);
        CoresAgentBusBegin.display(true);
        NagentByBus.display(true);
        std::cout << "N: " << Nagent << " B= " << Nbus << std::endl;*/
      
        
        //std::cout << " W0 : " << std::endl;
        //W0.display(true);
        calcW();
    
        dW.subtract(&W0, &W);
        A = MatrixGPU(B2, B2, 0, 1);
        P = MatrixGPU(B2 + 1, 1, 0, 1);
      
    }

   
        
    /*Ggrid2Bgrid2 = MatrixGPU(Nbus, Nbus);
    for (int i = 0; i < Nbus; i++) {
        for (int j = 0; j < Nbus; j++) {
            Ggrid2Bgrid2.set(i, j, sqrt(Ggrid.get(i, j) * Ggrid.get(i, j) + Bgrid.get(i, j) * Bgrid.get(i, j)));
        }
    }

    
    G = MatrixGPU(Nconstraint, N2);
    Phi = MatrixGPU(Nline, 1);
    Y = MatrixGPU(Nconstraint, 1);
    tempLN2 = MatrixGPU(Nline, N2);
    JacPhiE = MatrixGPU(Nline, B2);
    tempB2N2 = MatrixGPU(B2, N2);*/
    
    
#ifdef INSTRUMENTATION
    hipDeviceSynchronize();
    t2 = std::chrono::high_resolution_clock::now();
    timePerBlock.increment(0, 0, std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
    occurencePerBlock.increment(0, 0, 1);
#endif // INSTRUMENTATION

    //std::cout << " fin init" << std::endl;

}


void GPUPF::solve() {
  
    //std::cout << "solve Newton" << std::endl;
    time = clock();
    err = 2 * epsPF;
    iter = 0;
    int failure = 0;
    status = 1;
    //std::cout << epsPF << " " << iterM << std::endl;
    while (err > epsPF && iter < iterM) {
        
        failure = calcVoltage();
        if (failure) {
            status = -1;
            time = clock() - time;
            //std::cout << "failure ! " << iter << " " << err << std::endl;
            return;
        }
#ifdef INSTRUMENTATION
        t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION
        calcW();
#ifdef INSTRUMENTATION
        hipDeviceSynchronize();
        t2 = std::chrono::high_resolution_clock::now();
        timePerBlock.increment(0, 6, std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
        occurencePerBlock.increment(0, 6, 1);
        t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION
        if (_useDouble) {
            dWD.subtract(&W0D, &WD); // dW = W0 - W
            err = dWD.max2(); //err = ||dW||
        }
        else {
            dW.subtract(&W0, &W); // dW = W0 - W
            err = dW.max2(); //err = ||dW||
        }
#ifdef INSTRUMENTATION
        hipDeviceSynchronize();
        t2 = std::chrono::high_resolution_clock::now();
        timePerBlock.increment(0, 7, std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
        occurencePerBlock.increment(0, 7, 1);
#endif // INSTRUMENTATION      
        iter++;
        //std::cout << err << " * ";
    }
    //std::cout << std::endl;
#ifdef INSTRUMENTATION
    t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION
    calcW(true);
#ifdef INSTRUMENTATION
    hipDeviceSynchronize();
    t2 = std::chrono::high_resolution_clock::now();
    timePerBlock.increment(0, 6, std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
    occurencePerBlock.increment(0, 6, 1);
#endif // INSTRUMENTATION

    if (iter >= iterM) {
        status = 2;
        if (err > 100 * epsPF) {
            status = -1;
        }
        //std::cout << "fin solve " << iter<<" " << err << std::endl;
    }
    
    time = clock() - time;
        
}

void GPUPF::updatePQ(MatrixGPU* PQ)
{
#ifdef INSTRUMENTATION
    t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION
    calculW0Bis(PQ);
#ifdef INSTRUMENTATION
    t2 = std::chrono::high_resolution_clock::now();
    timePerBlock.increment(0, 8, std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
    occurencePerBlock.increment(0, 8, 1);
#endif
   

}

void GPUPF::calculW0(MatrixGPU* PQ)
{
    switch (_blockSize) {
    case 512:
        calcW0GPU<512> << <numBlock, _blockSize >> > (W0._matrixGPU, PQ->_matrixGPU, I._matrixGPU, Nagent, Nbus);
        break;
    case 256:
        calcW0GPU<256> << <numBlock, _blockSize >> > (W0._matrixGPU, PQ->_matrixGPU, I._matrixGPU, Nagent, Nbus);
        break;
    case 128:
        calcW0GPU<128> << <numBlock, _blockSize >> > (W0._matrixGPU, PQ->_matrixGPU, I._matrixGPU, Nagent, Nbus);
        break;
    case 64:
        calcW0GPU< 64> << <numBlock, _blockSize >> > (W0._matrixGPU, PQ->_matrixGPU, I._matrixGPU, Nagent, Nbus);
        break;
    case 32:
        calcW0GPU< 32> << <numBlock, _blockSize >> > (W0._matrixGPU, PQ->_matrixGPU, I._matrixGPU, Nagent, Nbus);
        break;
    case 16:
        calcW0GPU< 16> << <numBlock, _blockSize >> > (W0._matrixGPU, PQ->_matrixGPU, I._matrixGPU, Nagent, Nbus);
        break;
    case  8:
        calcW0GPU<  8> << <numBlock, _blockSize >> > (W0._matrixGPU, PQ->_matrixGPU, I._matrixGPU, Nagent, Nbus);
        break;
    case  4:
        calcW0GPU<  4> << <numBlock, _blockSize >> > (W0._matrixGPU, PQ->_matrixGPU, I._matrixGPU, Nagent, Nbus);
        break;
    case  2:
        calcW0GPU<  2> << <numBlock, _blockSize >> > (W0._matrixGPU, PQ->_matrixGPU, I._matrixGPU, Nagent, Nbus);
        break;
    case  1:
        calcW0GPU<  1> << <numBlock, _blockSize >> > (W0._matrixGPU, PQ->_matrixGPU, I._matrixGPU, Nagent, Nbus);
        break;
    }
}

void GPUPF::calculW0D(MatrixGPUD* PQD)
{
    switch (_blockSize) {
    case 512:
        calcW0GPUD<512> << <Nbus, _blockSize >> > (W0D._matrixGPU, PQD->_matrixGPU, I._matrixGPU, Nagent, Nbus);
        break;
    case 256:
        calcW0GPUD<256> << <Nbus, _blockSize >> > (W0D._matrixGPU, PQD->_matrixGPU, I._matrixGPU, Nagent, Nbus);
        break;
    case 128:
        calcW0GPUD<128> << <Nbus, _blockSize >> > (W0D._matrixGPU, PQD->_matrixGPU, I._matrixGPU, Nagent, Nbus);
        break;
    case 64:
        calcW0GPUD< 64> << <Nbus, _blockSize >> > (W0D._matrixGPU, PQD->_matrixGPU, I._matrixGPU, Nagent, Nbus);
        break;
    case 32:
        calcW0GPUD< 32> << <Nbus, _blockSize >> > (W0D._matrixGPU, PQD->_matrixGPU, I._matrixGPU, Nagent, Nbus);
        break;
    case 16:
        calcW0GPUD< 16> << <Nbus, _blockSize >> > (W0D._matrixGPU, PQD->_matrixGPU, I._matrixGPU, Nagent, Nbus);
        break;
    case  8:
        calcW0GPUD<  8> << <Nbus, _blockSize >> > (W0D._matrixGPU, PQD->_matrixGPU, I._matrixGPU, Nagent, Nbus);
        break;
    case  4:
        calcW0GPUD<  4> << <Nbus, _blockSize >> > (W0D._matrixGPU, PQD->_matrixGPU, I._matrixGPU, Nagent, Nbus);
        break;
    case  2:
        calcW0GPUD<  2> << <Nbus, _blockSize >> > (W0D._matrixGPU, PQD->_matrixGPU, I._matrixGPU, Nagent, Nbus);
        break;
    case  1:
        calcW0GPUD<  1> << <Nbus, _blockSize >> > (W0D._matrixGPU, PQD->_matrixGPU, I._matrixGPU, Nagent, Nbus);
        break;
    }
}

void GPUPF::calculW0Bis(MatrixGPU* PQ)
{
    // prend en compte le premier agent !!! 
    
    switch (_blockSize) {
    case 512:
        calcW0GPUBis<512> << <Nbus, _blockSize >> > (W0._matrixGPU, PQ->_matrixGPU, CoresAgentBus._matrixGPU, NagentByBus._matrixGPU, CoresAgentBusBegin._matrixGPU, Nagent, Nbus);
        break;
    case 256:
        calcW0GPUBis<256> << <Nbus, _blockSize >> > (W0._matrixGPU, PQ->_matrixGPU, CoresAgentBus._matrixGPU, NagentByBus._matrixGPU, CoresAgentBusBegin._matrixGPU, Nagent, Nbus);
        break;
    case 128:
        calcW0GPUBis<128> << <Nbus, _blockSize >> > (W0._matrixGPU, PQ->_matrixGPU, CoresAgentBus._matrixGPU, NagentByBus._matrixGPU, CoresAgentBusBegin._matrixGPU, Nagent, Nbus);
        break;
    case 64:
        calcW0GPUBis< 64> << <Nbus, _blockSize >> > (W0._matrixGPU, PQ->_matrixGPU, CoresAgentBus._matrixGPU, NagentByBus._matrixGPU, CoresAgentBusBegin._matrixGPU, Nagent, Nbus);
        break;
    case 32:
        calcW0GPUBis< 32> << <Nbus, _blockSize >> > (W0._matrixGPU, PQ->_matrixGPU, CoresAgentBus._matrixGPU, NagentByBus._matrixGPU, CoresAgentBusBegin._matrixGPU, Nagent, Nbus);
        break;
    case 16:
        calcW0GPUBis< 16> << <Nbus, _blockSize >> > (W0._matrixGPU, PQ->_matrixGPU, CoresAgentBus._matrixGPU, NagentByBus._matrixGPU, CoresAgentBusBegin._matrixGPU, Nagent, Nbus);
        break;
    case  8:
        calcW0GPUBis<  8> << <Nbus, _blockSize >> > (W0._matrixGPU, PQ->_matrixGPU, CoresAgentBus._matrixGPU, NagentByBus._matrixGPU, CoresAgentBusBegin._matrixGPU, Nagent, Nbus);
        break;
    case  4:
        calcW0GPUBis<  4> << <Nbus, _blockSize >> > (W0._matrixGPU, PQ->_matrixGPU, CoresAgentBus._matrixGPU, NagentByBus._matrixGPU, CoresAgentBusBegin._matrixGPU, Nagent, Nbus);
        break;
    case  2:
        calcW0GPUBis<  2> << <Nbus, _blockSize >> > (W0._matrixGPU, PQ->_matrixGPU, CoresAgentBus._matrixGPU, NagentByBus._matrixGPU, CoresAgentBusBegin._matrixGPU, Nagent, Nbus);
        break;
    case  1:
        calcW0GPUBis<  1> << <Nbus, _blockSize >> > (W0._matrixGPU, PQ->_matrixGPU, CoresAgentBus._matrixGPU, NagentByBus._matrixGPU, CoresAgentBusBegin._matrixGPU, Nagent, Nbus);
        break;
    }
}

void GPUPF::calculW0DBis(MatrixGPUD* PQD)
{
    switch (_blockSize) {
    case 512:
        calcW0GPUDBis<512> << <numBlock, _blockSize >> > (W0D._matrixGPU, PQD->_matrixGPU, CoresAgentBus._matrixGPU, NagentByBus._matrixGPU, CoresAgentBusBegin._matrixGPU, Nagent, Nbus);
        break;
    case 256:
        calcW0GPUDBis<256> << <numBlock, _blockSize >> > (W0D._matrixGPU, PQD->_matrixGPU, CoresAgentBus._matrixGPU, NagentByBus._matrixGPU, CoresAgentBusBegin._matrixGPU, Nagent, Nbus);
        break;
    case 128:
        calcW0GPUDBis<128> << <numBlock, _blockSize >> > (W0D._matrixGPU, PQD->_matrixGPU, CoresAgentBus._matrixGPU, NagentByBus._matrixGPU, CoresAgentBusBegin._matrixGPU, Nagent, Nbus);
        break;
    case 64:
        calcW0GPUDBis< 64> << <numBlock, _blockSize >> > (W0D._matrixGPU, PQD->_matrixGPU, CoresAgentBus._matrixGPU, NagentByBus._matrixGPU, CoresAgentBusBegin._matrixGPU, Nagent, Nbus);
        break;
    case 32:
        calcW0GPUDBis< 32> << <numBlock, _blockSize >> > (W0D._matrixGPU, PQD->_matrixGPU, CoresAgentBus._matrixGPU, NagentByBus._matrixGPU, CoresAgentBusBegin._matrixGPU, Nagent, Nbus);
        break;
    case 16:
        calcW0GPUDBis< 16> << <numBlock, _blockSize >> > (W0D._matrixGPU, PQD->_matrixGPU, CoresAgentBus._matrixGPU, NagentByBus._matrixGPU, CoresAgentBusBegin._matrixGPU, Nagent, Nbus);
        break;
    case  8:
        calcW0GPUDBis<  8> << <numBlock, _blockSize >> > (W0D._matrixGPU, PQD->_matrixGPU, CoresAgentBus._matrixGPU, NagentByBus._matrixGPU, CoresAgentBusBegin._matrixGPU, Nagent, Nbus);
        break;
    case  4:
        calcW0GPUDBis<  4> << <numBlock, _blockSize >> > (W0D._matrixGPU, PQD->_matrixGPU, CoresAgentBus._matrixGPU, NagentByBus._matrixGPU, CoresAgentBusBegin._matrixGPU, Nagent, Nbus);
        break;
    case  2:
        calcW0GPUDBis<  2> << <numBlock, _blockSize >> > (W0D._matrixGPU, PQD->_matrixGPU, CoresAgentBus._matrixGPU, NagentByBus._matrixGPU, CoresAgentBusBegin._matrixGPU, Nagent, Nbus);
        break;
    case  1:
        calcW0GPUDBis<  1> << <numBlock, _blockSize >> > (W0D._matrixGPU, PQD->_matrixGPU, CoresAgentBus._matrixGPU, NagentByBus._matrixGPU, CoresAgentBusBegin._matrixGPU, Nagent, Nbus);
        break;
    }
}


void GPUPF::calcW(bool end)
{
    /*
    ED.display(true);
        _GlinD.display(true);
        _BlinD.display(true); 
        std::cout << "------------" << std::endl;
        _CoresVoiLin.display(true);
        _CoresBusLin.display(true);
        _nLines.display(true);
        std::cout << "------------" << std::endl;
    _PintermediateD.display(true);
        _QintermediateD.display(true);
    _Qintermediate.display(true);
    std::cout << "------------" << std::endl;*/ 
    if (_useDouble) {
        
        calcWinterD << <numBlock, _blockSize, B2 * sizeof(double) >> > (_PintermediateD._matrixGPU, _QintermediateD._matrixGPU, ED._matrixGPU, _GlinD._matrixGPU, _BlinD._matrixGPU, _CoresVoiLin._matrixGPU, _CoresBusLin._matrixGPU, _nLines._matrixGPU, Nbus);
        
        if (!end) { // pendant simu, la puissance � ce noeud est libre
            switch (_blockSize) {
            case 512:
                calcWGPUD<512> << <numBlock, _blockSize >> > (WD._matrixGPU, W0D._matrixGPU, _PintermediateD._matrixGPU, _QintermediateD._matrixGPU, _CoresBusLin._matrixGPU, _nLines._matrixGPU, Nbus);
                break;
            case 256:
                calcWGPUD<256> << <numBlock, _blockSize >> > (WD._matrixGPU, W0D._matrixGPU, _PintermediateD._matrixGPU, _QintermediateD._matrixGPU, _CoresBusLin._matrixGPU, _nLines._matrixGPU, Nbus);
                break;
            case 128:
                calcWGPUD<128> << <numBlock, _blockSize >> > (WD._matrixGPU, W0D._matrixGPU, _PintermediateD._matrixGPU, _QintermediateD._matrixGPU, _CoresBusLin._matrixGPU, _nLines._matrixGPU, Nbus);
                break;
            case 64:
                calcWGPUD< 64> << <numBlock, _blockSize >> > (WD._matrixGPU, W0D._matrixGPU, _PintermediateD._matrixGPU, _QintermediateD._matrixGPU, _CoresBusLin._matrixGPU, _nLines._matrixGPU, Nbus);
                break;
            case 32:
                calcWGPUD< 32> << <numBlock, _blockSize >> > (WD._matrixGPU, W0D._matrixGPU, _PintermediateD._matrixGPU, _QintermediateD._matrixGPU, _CoresBusLin._matrixGPU, _nLines._matrixGPU, Nbus);
                break;
            case 16:
                calcWGPUD< 16> << <numBlock, _blockSize >> > (WD._matrixGPU, W0D._matrixGPU, _PintermediateD._matrixGPU, _QintermediateD._matrixGPU, _CoresBusLin._matrixGPU, _nLines._matrixGPU, Nbus);
                break;
            case  8:
                calcWGPUD<  8> << <numBlock, _blockSize >> > (WD._matrixGPU, W0D._matrixGPU, _PintermediateD._matrixGPU, _QintermediateD._matrixGPU, _CoresBusLin._matrixGPU, _nLines._matrixGPU, Nbus);
                break;
            case  4:
                calcWGPUD<  4> << <numBlock, _blockSize >> > (WD._matrixGPU, W0D._matrixGPU, _PintermediateD._matrixGPU, _QintermediateD._matrixGPU, _CoresBusLin._matrixGPU, _nLines._matrixGPU, Nbus);
                break;
            case  2:
                calcWGPUD<  2> << <numBlock, _blockSize >> > (WD._matrixGPU, W0D._matrixGPU, _PintermediateD._matrixGPU, _QintermediateD._matrixGPU, _CoresBusLin._matrixGPU, _nLines._matrixGPU, Nbus);
                break;
            case  1:
                calcWGPUD<  1> << <numBlock, _blockSize >> > (WD._matrixGPU, W0D._matrixGPU, _PintermediateD._matrixGPU, _QintermediateD._matrixGPU, _CoresBusLin._matrixGPU, _nLines._matrixGPU, Nbus);
                break;
            }
        }
        else {
            switch (_blockSize) {
            case 512:
                calcWGPUD<512> << <numBlock, _blockSize >> > (WD._matrixGPU, _PintermediateD._matrixGPU, _QintermediateD._matrixGPU, _CoresBusLin._matrixGPU, _nLines._matrixGPU, Nbus);
                break;
            case 256:
                calcWGPUD<256> << <numBlock, _blockSize >> > (WD._matrixGPU, _PintermediateD._matrixGPU, _QintermediateD._matrixGPU, _CoresBusLin._matrixGPU, _nLines._matrixGPU, Nbus);
                break;
            case 128:
                calcWGPUD<128> << <numBlock, _blockSize >> > (WD._matrixGPU, _PintermediateD._matrixGPU, _QintermediateD._matrixGPU, _CoresBusLin._matrixGPU, _nLines._matrixGPU, Nbus);
                break;
            case 64:
                calcWGPUD< 64> << <numBlock, _blockSize >> > (WD._matrixGPU, _PintermediateD._matrixGPU, _QintermediateD._matrixGPU, _CoresBusLin._matrixGPU, _nLines._matrixGPU, Nbus);
                break;
            case 32:
                calcWGPUD< 32> << <numBlock, _blockSize >> > (WD._matrixGPU, _PintermediateD._matrixGPU, _QintermediateD._matrixGPU, _CoresBusLin._matrixGPU, _nLines._matrixGPU, Nbus);
                break;
            case 16:
                calcWGPUD< 16> << <numBlock, _blockSize >> > (WD._matrixGPU, _PintermediateD._matrixGPU, _QintermediateD._matrixGPU, _CoresBusLin._matrixGPU, _nLines._matrixGPU, Nbus);
                break;
            case  8:
                calcWGPUD<  8> << <numBlock, _blockSize >> > (WD._matrixGPU, _PintermediateD._matrixGPU, _QintermediateD._matrixGPU, _CoresBusLin._matrixGPU, _nLines._matrixGPU, Nbus);
                break;
            case  4:
                calcWGPUD<  4> << <numBlock, _blockSize >> > (WD._matrixGPU, _PintermediateD._matrixGPU, _QintermediateD._matrixGPU, _CoresBusLin._matrixGPU, _nLines._matrixGPU, Nbus);
                break;
            case  2:
                calcWGPUD<  2> << <numBlock, _blockSize >> > (WD._matrixGPU, _PintermediateD._matrixGPU, _QintermediateD._matrixGPU, _CoresBusLin._matrixGPU, _nLines._matrixGPU, Nbus);
                break;
            case  1:
                calcWGPUD<  1> << <numBlock, _blockSize >> > (WD._matrixGPU, _PintermediateD._matrixGPU, _QintermediateD._matrixGPU, _CoresBusLin._matrixGPU, _nLines._matrixGPU, Nbus);
                break;
            }

        }
    }
    else {
       /* E.display(true);
        _Glin.display(true);
        _Blin.display(true);
        std::cout << "------------" << std::endl;
        _CoresVoiLin.display(true);
        _CoresBusLin.display(true);
        _nLines.display(true);
        std::cout << "------------" << std::endl;*/

        calcWinter << <numBlock, _blockSize, B2 * sizeof(float) >> > (_Pintermediate._matrixGPU, _Qintermediate._matrixGPU, E._matrixGPU, _Glin._matrixGPU, _Blin._matrixGPU, _CoresVoiLin._matrixGPU, _CoresBusLin._matrixGPU, _nLines._matrixGPU, Nbus);

        /*_Pintermediate.display(true);
        _Qintermediate.display(true);*/
        if (!end) { // pendant simu, la puissance � ce noeud est libre
            switch (_blockSize) {
            case 512:
                calcWGPU<512> << <numBlock, _blockSize >> > (W._matrixGPU, W0._matrixGPU, _Pintermediate._matrixGPU, _Qintermediate._matrixGPU, _CoresBusLin._matrixGPU, _nLines._matrixGPU, Nbus);
                break;
            case 256:
                calcWGPU<256> << <numBlock, _blockSize >> > (W._matrixGPU, W0._matrixGPU, _Pintermediate._matrixGPU, _Qintermediate._matrixGPU, _CoresBusLin._matrixGPU, _nLines._matrixGPU, Nbus);
                break;
            case 128:
                calcWGPU<128> << <numBlock, _blockSize >> > (W._matrixGPU, W0._matrixGPU, _Pintermediate._matrixGPU, _Qintermediate._matrixGPU, _CoresBusLin._matrixGPU, _nLines._matrixGPU, Nbus);
                break;
            case 64:
                calcWGPU< 64> << <numBlock, _blockSize >> > (W._matrixGPU, W0._matrixGPU, _Pintermediate._matrixGPU, _Qintermediate._matrixGPU, _CoresBusLin._matrixGPU, _nLines._matrixGPU, Nbus);
                break;
            case 32:
                calcWGPU< 32> << <numBlock, _blockSize >> > (W._matrixGPU, W0._matrixGPU, _Pintermediate._matrixGPU, _Qintermediate._matrixGPU, _CoresBusLin._matrixGPU, _nLines._matrixGPU, Nbus);
                break;
            case 16:
                calcWGPU< 16> << <numBlock, _blockSize >> > (W._matrixGPU, W0._matrixGPU, _Pintermediate._matrixGPU, _Qintermediate._matrixGPU, _CoresBusLin._matrixGPU, _nLines._matrixGPU, Nbus);
                break;
            case  8:
                calcWGPU<  8> << <numBlock, _blockSize >> > (W._matrixGPU, W0._matrixGPU, _Pintermediate._matrixGPU, _Qintermediate._matrixGPU, _CoresBusLin._matrixGPU, _nLines._matrixGPU, Nbus);
                break;
            case  4:
                calcWGPU<  4> << <numBlock, _blockSize >> > (W._matrixGPU, W0._matrixGPU, _Pintermediate._matrixGPU, _Qintermediate._matrixGPU, _CoresBusLin._matrixGPU, _nLines._matrixGPU, Nbus);
                break;
            case  2:
                calcWGPU<  2> << <numBlock, _blockSize >> > (W._matrixGPU, W0._matrixGPU, _Pintermediate._matrixGPU, _Qintermediate._matrixGPU, _CoresBusLin._matrixGPU, _nLines._matrixGPU, Nbus);
                break;
            case  1:
                calcWGPU<  1> << <numBlock, _blockSize >> > (W._matrixGPU, W0._matrixGPU, _Pintermediate._matrixGPU, _Qintermediate._matrixGPU, _CoresBusLin._matrixGPU, _nLines._matrixGPU, Nbus);
                break;
            }
        }
        else {
            switch (_blockSize) {
            case 512:
                calcWGPU<512> << <numBlock, _blockSize >> > (W._matrixGPU, _Pintermediate._matrixGPU, _Qintermediate._matrixGPU, _CoresBusLin._matrixGPU, _nLines._matrixGPU, Nbus);
                break;
            case 256:
                calcWGPU<256> << <numBlock, _blockSize >> > (W._matrixGPU, _Pintermediate._matrixGPU, _Qintermediate._matrixGPU, _CoresBusLin._matrixGPU, _nLines._matrixGPU, Nbus);
                break;
            case 128:
                calcWGPU<128> << <numBlock, _blockSize >> > (W._matrixGPU, _Pintermediate._matrixGPU, _Qintermediate._matrixGPU, _CoresBusLin._matrixGPU, _nLines._matrixGPU, Nbus);
                break;
            case 64:
                calcWGPU< 64> << <numBlock, _blockSize >> > (W._matrixGPU, _Pintermediate._matrixGPU, _Qintermediate._matrixGPU, _CoresBusLin._matrixGPU, _nLines._matrixGPU, Nbus);
                break;
            case 32:
                calcWGPU< 32> << <numBlock, _blockSize >> > (W._matrixGPU, _Pintermediate._matrixGPU, _Qintermediate._matrixGPU, _CoresBusLin._matrixGPU, _nLines._matrixGPU, Nbus);
                break;
            case 16:
                calcWGPU< 16> << <numBlock, _blockSize >> > (W._matrixGPU, _Pintermediate._matrixGPU, _Qintermediate._matrixGPU, _CoresBusLin._matrixGPU, _nLines._matrixGPU, Nbus);
                break;
            case  8:
                calcWGPU<  8> << <numBlock, _blockSize >> > (W._matrixGPU, _Pintermediate._matrixGPU, _Qintermediate._matrixGPU, _CoresBusLin._matrixGPU, _nLines._matrixGPU, Nbus);
                break;
            case  4:
                calcWGPU<  4> << <numBlock, _blockSize >> > (W._matrixGPU, _Pintermediate._matrixGPU, _Qintermediate._matrixGPU, _CoresBusLin._matrixGPU, _nLines._matrixGPU, Nbus);
                break;
            case  2:
                calcWGPU<  2> << <numBlock, _blockSize >> > (W._matrixGPU, _Pintermediate._matrixGPU, _Qintermediate._matrixGPU, _CoresBusLin._matrixGPU, _nLines._matrixGPU, Nbus);
                break;
            case  1:
                calcWGPU<  1> << <numBlock, _blockSize >> > (W._matrixGPU, _Pintermediate._matrixGPU, _Qintermediate._matrixGPU, _CoresBusLin._matrixGPU, _nLines._matrixGPU, Nbus);
                break;
            }

        }

    }
   
    //W.display(true);
    
}

void GPUPF::calcJac()
{
    if (_useDouble) {
        calcJacGPUD <<<numBlock, _blockSize, B2 * sizeof(double) >>> (JacD._matrixGPU, WD._matrixGPU, ED._matrixGPU, _GlinD._matrixGPU, _BlinD._matrixGPU, _CoresVoiLin._matrixGPU, _CoresBusLin._matrixGPU, _nLines._matrixGPU, Nbus);
    }
    else {
        calcJacGPU <<<numBlock, _blockSize,  B2 * sizeof(float) >>> (Jac._matrixGPU, W._matrixGPU, E._matrixGPU, _Glin._matrixGPU, _Blin._matrixGPU, _CoresVoiLin._matrixGPU, _CoresBusLin._matrixGPU, _nLines._matrixGPU, Nbus);
    }
}

void GPUPF::calcPhi()
{
    calcE();
    
    calculPhiGPU << <numBlock, _blockSize >> > (Phi._matrixGPU, E._matrixGPU, _Blin2._matrixGPU, _Glin2._matrixGPU, CoresLineBusGPU._matrixGPU, Nbus, Nline);
}

void GPUPF::calcJacPhiE()
{
    for (int l = 0; l < Nline; l++) { //angle
        int i = CoresLineBus.get(l, 0); //from 
        int i2 = i + Nbus;
        int j = CoresLineBus.get(l, 1); // to
        int j2 = j + Nbus;
        
        JacPhiE.set(l, i2, E.get(j2, 0) * Ggrid2Bgrid2.get(i, j));
        JacPhiE.set(l, j2, E.get(i2, 0) * Ggrid2Bgrid2.get(i, j));

        /*float dTheta_ij = E.get(i, 0) - E.get(j, 0);

        JacPhiE.set(l, i, -E.get(i2, 0) * E.get(j2, 0) * (Ggrid.get(i, j) * sin(dTheta_ij) - Bgrid.get(i, j) * cos(dTheta_ij)));
        JacPhiE.set(l, j,  E.get(i2, 0) * E.get(j2, 0) * (Ggrid.get(i, j) * sin(dTheta_ij) - Bgrid.get(i, j) * cos(dTheta_ij)));
        JacPhiE.set(l, i2, E.get(j2, 0) *                (Ggrid.get(i, j) * cos(dTheta_ij) + Bgrid.get(i, j) * sin(dTheta_ij)));
        JacPhiE.set(l, j2, E.get(i2, 0) *                (Ggrid.get(i, j) * cos(dTheta_ij) + Bgrid.get(i, j) * sin(dTheta_ij)));
        */
    }

}

int GPUPF::calcVoltage()
{
#ifdef INSTRUMENTATION
    t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION
    calcJac();
#ifdef INSTRUMENTATION
    hipDeviceSynchronize();
    t2 = std::chrono::high_resolution_clock::now();
    timePerBlock.increment(0, 3, std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
    occurencePerBlock.increment(0, 3, 1);
#endif // INSTRUMENTATION
    


    if (_useDouble) {
#ifdef INSTRUMENTATION
        t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION
        try
        {
            JacD.LUPFactorization(&AD, &PD);
        }
        catch (const std::exception&)
        {
            return 1; // failure
        }
        
        /*AD.display(true);
        PD.display(true);
         dWD.display(true);*/
#ifdef INSTRUMENTATION
        hipDeviceSynchronize();
        t2 = std::chrono::high_resolution_clock::now();
        timePerBlock.increment(0, 4, std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
        occurencePerBlock.increment(0, 4, 1);
#endif // INSTRUMENTATION

        dED.solveSys(&AD, &PD, &dWD);
#ifdef INSTRUMENTATION
        hipDeviceSynchronize();
        t2 = std::chrono::high_resolution_clock::now();
        timePerBlock.increment(0, 5, std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
        occurencePerBlock.increment(0, 5, 1);
#endif // INSTRUMENTATION
        //dED.display(true);
        //std::cout << "**********" << std::endl;
       
       /*try
        {
            JacInvD.invertGaussJordan(&JacD);
        }
        catch (const std::exception&)
        {
            JacD.display(true);
            exit(0);
        }
        dED.multiply(&JacInvD, &dWD);*//**/// dE = Jac_inv * dW;
        ED.add(&ED, &dED);// E = E + dE;/*
    }
    else {
        //std::cout << " Jac : " << std::endl;
        /*try
        {
            JacInv.invertGaussJordan(&Jac);
        }
        catch (const std::exception&)
        {
            exit(0);
        }
        dE.multiply(&JacInv, &dW);*/// dE = Jac_inv * dW;
#ifdef INSTRUMENTATION
        t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION
        try
        {
            Jac.LUPFactorization(&A, &P);
        }
        catch (const std::exception&)
        {
            return 1;
        }
#ifdef INSTRUMENTATION
        hipDeviceSynchronize();
        t2 = std::chrono::high_resolution_clock::now();
        timePerBlock.increment(0, 4, std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
        occurencePerBlock.increment(0, 4, 1);
#endif // INSTRUMENTATION
        dE.solveSys(&A, &P, &dW);/**/
#ifdef INSTRUMENTATION
        hipDeviceSynchronize();
        t2 = std::chrono::high_resolution_clock::now();
        timePerBlock.increment(0, 5, std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
        occurencePerBlock.increment(0, 5, 1);
#endif // INSTRUMENTATION       
        E.add(&E, &dE);// E = E + dE;
    }
    return 0;
}

void GPUPF::calcE()
{
    // nothing to do
}

MatrixGPU* GPUPF::calcG()
{
    calcJacPhiE();
   
    tempB2N2.multiplyMat(&JacInv, &I_aug);
    
    tempLN2.multiplyMat(&JacPhiE, &tempB2N2);
    
    G.setBloc(0, B2, 0, N2, &tempB2N2);
    
    G.setBloc(B2, Nconstraint, 0, N2, &tempLN2);

    return &G;
}

MatrixGPU GPUPF::getY()
{
    //std::cout << "getY GPUPF" << std::endl;
    //CHECK_LAST_CUDA_ERROR();
    // E.display(true);
    calcPhi();
    //Phi.display();
    //CHECK_LAST_CUDA_ERROR();
    setY << <numBlock, _blockSize >> > (Y._matrixGPU, E._matrixGPU, Phi._matrixGPU, Nbus, Nline);
    //CHECK_LAST_CUDA_ERROR();
    //Y.display(true);
    return Y;
}

void GPUPF::setE(MatrixGPU* Enew)
{
#ifdef INSTRUMENTATION
    t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION
    E = *Enew;
    if (!E.getPos()) {
        E.transferGPU();
    }
    if (_useDouble) {
        E.toMatGPUD(ED);
    }
#ifdef INSTRUMENTATION
    t2 = std::chrono::high_resolution_clock::now();
    timePerBlock.increment(0, 8, std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
    occurencePerBlock.increment(0, 8, 1);
    t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION
    calcW();
#ifdef INSTRUMENTATION
    t2 = std::chrono::high_resolution_clock::now();
    timePerBlock.increment(0, 6, std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
    occurencePerBlock.increment(0, 6, 1);
    t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION
    if (_useDouble) {
        dWD.subtract(&W0D, &WD);
        err = dWD.max2(); //err = ||dW||
    }
    else {
        dW.subtract(&W0, &W);
        err = dW.max2(); //err = ||dW||
    }
#ifdef INSTRUMENTATION
    t2 = std::chrono::high_resolution_clock::now();
    timePerBlock.increment(0, 7, std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
    occurencePerBlock.increment(0, 7, 1);
#endif // INSTRUMENTATION


    
    
}

void GPUPF::setE(MatrixGPUD* Enew)
{
#ifdef INSTRUMENTATION
    t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION
    ED = *Enew;    
    if (!ED.getPos()) {
        ED.transferGPU();
    }
    if (!_useDouble) {
        E = ED;
    }
#ifdef INSTRUMENTATION
    hipDeviceSynchronize();
    t2 = std::chrono::high_resolution_clock::now();
    timePerBlock.increment(0, 8, std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
    occurencePerBlock.increment(0, 8, 1);
    t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION
    calcW();
#ifdef INSTRUMENTATION
    hipDeviceSynchronize();
    t2 = std::chrono::high_resolution_clock::now();
    timePerBlock.increment(0, 6, std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
    occurencePerBlock.increment(0, 6, 1);
    t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION
    if (_useDouble) {
        dWD.subtract(&W0D, &WD);
        err = dWD.max2(); //err = ||dW||
    }
    else {
        dW.subtract(&W0, &W);
        err = dW.max2(); //err = ||dW||
    }
#ifdef INSTRUMENTATION
    hipDeviceSynchronize();
    t2 = std::chrono::high_resolution_clock::now();
    timePerBlock.increment(0, 7, std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
    occurencePerBlock.increment(0, 7, 1);
#endif // INSTRUMENTATION
}

void GPUPF::setW(MatrixGPU* Wnew)
{
    W = *Wnew;
}

float GPUPF::getPloss()
{
    float s = 0;
    
    if (_useDouble) {
        s = WD.sum(0, Nbus);
    }
    else {
        s = W.sum(0, Nbus);
    }
   
    return s; // desequilibre !
}

float GPUPF::getQloss()
{
    float s = 0;
    
    if (_useDouble) {
        s = WD.sum(Nbus, B2);

    }
    else {
        s = W.sum(Nbus, B2);
    }
    
    return s; // desequilibre !
}

float GPUPF::getRes()
{
    return err;
}

int GPUPF::getIter()
{
    return iter;
}

float GPUPF::getP0()
{
    if (_useDouble) {
        return WD.get(0, 0, false);
    }
    else {
        return W.get(0, 0, false);
    }
}

float GPUPF::getQ0()
{
    if (_useDouble) {
        return WD.get(Nbus, 0, false);
    }
    else {
        return W.get(Nbus, 0, false);
    }
}

float GPUPF::getTime()
{
    return (float)time / CLOCKS_PER_SEC;
}

int GPUPF::getConv()
{
    return status;
}

MatrixCPU GPUPF::getE()
{
    if (_useDouble) {
        if(ED.max2()==0){
            calcE();
        }
        E = ED;
    }
    if(E.max2()==0){
        calcE();
    }
    MatrixCPU ECPU;
    E.toMatCPU(ECPU);

    return ECPU;
}
MatrixCPU GPUPF::getW()
{
    if (_useDouble) {
        if (WD.get(0,0)== 0)
        {
           calcW(true);
        }
        W = WD;
    }
    if(W.get(0,0)== 0){
        calcW(true);
    }

    MatrixCPU WCPU;
    W.toMatCPU(WCPU);

    return WCPU;
}



void GPUPF::display()
{
    std::cout << "-----------Resultat du PF --------" << std::endl;

    std::cout << "Nombre d'iter " << iter << " precision atteinte " << err << " temps de resolution " << (float)time / CLOCKS_PER_SEC <<  std::endl;
    //std::cout << " Puissance d'entree" << std::endl;
    //W0.display();
    std::cout << " Puissance active-reactive" << std::endl;
    W.display();
    std::cout << " Tension angle-tension" << std::endl;
    E.display();

    
    std::cout << "Pertes actives  " << getPloss() << std::endl;
    std::cout << "Pertes reactive " << getQloss() << std::endl;
    std::cout << "---------------------------" << std::endl;
}

void GPUPF::display2(bool all)
{
    std::cout.precision(3);

    if (_useDouble) {
        WD.transferCPU();
        W0D.transferCPU();
        dWD.transferCPU();
        ED.transferCPU();
    }
    else {
        W.transferCPU();
        W0.transferCPU();
        dW.transferCPU();
        E.transferCPU();
    }


    if (iter == 0) {
        std::cout << "algorithm not launch" << std::endl;
        if (_useDouble) {
            double temp = WD.get(0, 0);
            double temp2 = WD.get(Nbus, 0);
            WD.set(0, 0, W0.get(0, 0));
            WD.set(Nbus, 0, W0.get(Nbus, 0));
            dWD.subtract(&W0D, &WD); // dW = W0 - W
            err = dWD.max2(); //err = ||dW||
            /*for (int b = 0; b < Nbus; b++) {
                std::cout << b << " " << dW.get(b, 0) << " " << dW.get(Nbus + b, 0) << std::endl;
            }*/
            WD.set(0, 0, temp);
            WD.set(Nbus, 0, temp2);
        }
        else {
            float temp = W.get(0, 0);
            float temp2 = W.get(Nbus, 0);
            W.set(0, 0, W0.get(0, 0));
            W.set(Nbus, 0, W0.get(Nbus, 0));
            dW.subtract(&W0, &W); // dW = W0 - W
            err = dW.max2(); //err = ||dW||
            /*for (int b = 0; b < Nbus; b++) {
                std::cout << b << " " << dW.get(b, 0) << " " << dW.get(Nbus + b, 0) << std::endl;
            }*/
            W.set(0, 0, temp);
            W.set(Nbus, 0, temp2);
        }
    }
    else if (iter < iterM) {
        std::cout << "method " << _name << " on GPU converged in " << iter << " iterations." << std::endl;
        std::cout << "Converged in " << (float)time / CLOCKS_PER_SEC << " seconds" << std::endl;
        if (_useDouble) {
            std::cout << " Computation with double precision" << std::endl;
            double temp = WD.get(0, 0);
            double temp2 = WD.get(Nbus, 0);
            WD.set(0, 0, W0D.get(0, 0));
            WD.set(Nbus, 0, W0D.get(Nbus, 0));
            dWD.subtract(&W0D, &WD); // dW = W0 - W
            err = dWD.max2(); //err = ||dW||
            /*for (int b = 0; b < Nbus; b++) {
                std::cout << b << " " << dW.get(b, 0) << " " << dW.get(Nbus + b, 0) << std::endl;
            }*/
            WD.set(0, 0, temp);
            WD.set(Nbus, 0, temp2);
        }
        else {
            std::cout << " Computation with float simple precision" << std::endl;
            float temp = W.get(0, 0);
            float temp2 = W.get(Nbus, 0);
            W.set(0, 0, W0.get(0, 0));
            W.set(Nbus, 0, W0.get(Nbus, 0));
            dW.subtract(&W0, &W); // dW = W0 - W
            err = dW.max2(); //err = ||dW||
            /*for (int b = 0; b < Nbus; b++) {
                std::cout << b << " " << dW.get(b, 0) << " " << dW.get(Nbus + b, 0) << std::endl;
            }*/
            W.set(0, 0, temp);
            W.set(Nbus, 0, temp2);
        }
    }
    else {
        std::cout << "method " << _name << " on GPU not converged in " << iter << " iterations." << std::endl;
        std::cout << "time taken " << (float)time / CLOCKS_PER_SEC << " seconds" << std::endl;
        if (_useDouble) {
            std::cout << " Computation with double precision" << std::endl;
        }
        else {
            std::cout << " Computation with float simple precision, maibe try with double to converge" << std::endl;
        }
    }
    std::cout << "The power error of this state is " << err << std::endl;
    std::cout << "===============================================================|" << std::endl;
    std::cout << "      System Summary                                           |" << std::endl;
    std::cout << "===============================================================|" << std::endl;
    std::cout << "Buses            " << Nbus << std::endl;
    std::cout << "Branches         " << Nline << std::endl;
    std::cout << "Ploss            " << getPloss() << std::endl;
    std::cout << "Qloss            " << getQloss() << std::endl;


    std::cout << std::endl << std::endl;
    std::cout << "===============================================================================================|" << std::endl;
    std::cout << "      Bus Data                                                                                 |" << std::endl;
    std::cout << "===============================================================================================|" << std::endl;
    std::cout << " Bus |          Voltage        |  Power = Generation  + Load   |  Init = Generation  + Load    |" << std::endl;
    std::cout << "  #  |    Mag(pu) |  Ang(deg)  |    P (pu)     |     Q (pu)    |    P (pu)     |     Q (pu)    |" << std::endl;
    std::cout << "-----|------------|------------|---------------|---------------|---------------|---------------|" << std::endl;
    float seuil = 0.0001;
    //std::cout << 0 << "      " << E.get(Nbus, 0) << "             " << E.get(0, 0) * (abs(E.get(0, 0)) > 0.0001) * 180 / 3.1415 << "              " << (abs(W.get(0, 0)) > 0.0001) * W.get(0, 0) << "         " << (abs(W.get(Nbus, 0)) > 0.0001) * W.get(Nbus, 0) << std::endl;
    //ED.display(true);
    if (all) {
        if (_useDouble) {
            std::cout << std::setw(5) << 0 << "|" << std::setw(11) << ED.get(Nbus, 0) << "*|" << std::setw(11) << ED.get(0, 0) * (abs(ED.get(0, 0)) > seuil) * 180 / 3.1415
                << "*|" << std::setw(15) << (abs(WD.get(0, 0)) > seuil) * WD.get(0, 0) << "|" << std::setw(15) << (abs(WD.get(Nbus, 0)) > seuil) * WD.get(Nbus, 0)
                << "|" << std::setw(15) << W0D.get(0, 0) << "|" << std::setw(15)
                << W0D.get(Nbus, 0) << "|" << std::endl;
            for (int b = 1; b < Nbus; b++) {
                //std::cout.width(10);
                //std::cout << b << "      " << E.get(b + Nbus, 0) << "        " << E.get(b, 0) * (abs(E.get(b, 0)) > 0.0001) * 180 / 3.1415 << "          " << (abs(W.get(b, 0)) > 0.0001) * W.get(b, 0) << "         " << (abs(W.get(b + Nbus, 0)) > 0.0001) * W.get(b + Nbus, 0) << std::endl;
                std::cout << std::setw(5) << b << "|" << std::setw(11) << ED.get(b + Nbus, 0) << " |" << std::setw(11)
                    << ED.get(b, 0) * (abs(ED.get(b, 0)) > seuil) * 180 / 3.1415 << " |" << std::setw(15)
                    << (abs(WD.get(b, 0)) > seuil) * WD.get(b, 0) << "|" << std::setw(15) << (abs(WD.get(b + Nbus, 0)) > seuil) * WD.get(b + Nbus, 0)
                    << "|" << std::setw(15) << (abs(W0D.get(b, 0)) > seuil) * W0D.get(b, 0) << "|" << std::setw(15)
                    << W0D.get(b + Nbus, 0) << "|" << std::endl;

            }
        }
        else {
            std::cout << std::setw(5) << 0 << "|" << std::setw(11) << E.get(Nbus, 0) << "*|" << std::setw(11) << E.get(0, 0) * (abs(E.get(0, 0)) > seuil) * 180 / 3.1415
                << "*|" << std::setw(15) << (abs(W.get(0, 0)) > seuil) * W.get(0, 0) << "|" << std::setw(15) << (abs(W.get(Nbus, 0)) > seuil) * W.get(Nbus, 0)
                << "|" << std::setw(15) << W0.get(0, 0) << "|" << std::setw(15)
                << W0.get(Nbus, 0) << "|" << std::endl;
            for (int b = 1; b < Nbus; b++) {
                //std::cout.width(10);
                //std::cout << b << "      " << E.get(b + Nbus, 0) << "        " << E.get(b, 0) * (abs(E.get(b, 0)) > 0.0001) * 180 / 3.1415 << "          " << (abs(W.get(b, 0)) > 0.0001) * W.get(b, 0) << "         " << (abs(W.get(b + Nbus, 0)) > 0.0001) * W.get(b + Nbus, 0) << std::endl;
                std::cout << std::setw(5) << b << "|" << std::setw(11) << E.get(b + Nbus, 0) << " |" << std::setw(11)
                    << E.get(b, 0) * (abs(E.get(b, 0)) > seuil) * 180 / 3.1415 << " |" << std::setw(15)
                    << (abs(W.get(b, 0)) > seuil) * W.get(b, 0) << "|" << std::setw(15) << (abs(W.get(b + Nbus, 0)) > seuil) * W.get(b + Nbus, 0)
                    << "|" << std::setw(15) << W0.get(b, 0) << "|" << std::setw(15)
                    << W0.get(b + Nbus, 0) << "|" << std::endl;

            }
        }
    }
    else {
        if (_useDouble) {
            std::cout << std::setw(5) << 0 << "|" << std::setw(11) << ED.get(Nbus, 0) << "*|" << std::setw(11) << ED.get(0, 0) * (abs(ED.get(0, 0)) > seuil) * 180 / 3.1415
                << "*|" << std::setw(15) << (abs(WD.get(0, 0)) > seuil) * WD.get(0, 0) << "|" << std::setw(15) << (abs(WD.get(Nbus, 0)) > seuil) * WD.get(Nbus, 0)
                << "|" << std::setw(15) << W0D.get(0, 0) << "|" << std::setw(15)
                << W0D.get(Nbus, 0) << "|" << std::endl;
        }
        else {
            std::cout << std::setw(5) << 0 << "|" << std::setw(11) << E.get(Nbus, 0) << "*|" << std::setw(11) << E.get(0, 0) * (abs(E.get(0, 0)) > seuil) * 180 / 3.1415
                << "*|" << std::setw(15) << (abs(W.get(0, 0)) > seuil) * W.get(0, 0) << "|" << std::setw(15) << (abs(W.get(Nbus, 0)) > seuil) * W.get(Nbus, 0)
                << "|" << std::setw(15) << W0.get(0, 0) << "|" << std::setw(15)
                << W0.get(Nbus, 0) << "|" << std::endl;
        }
    }




    std::cout << "===============================================================================================|" << std::endl;
    std::cout << "                      END PRINT                                                                |" << std::endl;
    std::cout << "===============================================================================================|" << std::endl;

}

void GPUPF::saveTimeBlock(std::string fileName)
{
    std::ios_base::openmode mode = std::fstream::in | std::fstream::out | std::fstream::app;
    float factor = 1000000; // go from ns to ms fot the printed time


    if (occurencePerBlock.get(0, 0) != 0) {
        std::cout << "total resolution time :" << timePerBlock.sum() / (1000 * factor) << "s" << std::endl;
        std::cout << " Fb0 : " << timePerBlock.get(0, 0) / factor << "ms and occurence :" << occurencePerBlock.get(0, 0) << std::endl;
        if (occurencePerBlock.get(0, 3) != 0) {
            std::cout << " Fb1a : " << timePerBlock.get(0, 1) / factor << "ms and occurence :" << occurencePerBlock.get(0, 1) << std::endl;
            std::cout << " Fb1b : " << timePerBlock.get(0, 2) / factor << "ms and occurence :" << occurencePerBlock.get(0, 2) << std::endl;
            std::cout << " Fb1c : " << timePerBlock.get(0, 3) / factor << "ms and occurence :" << occurencePerBlock.get(0, 3) << std::endl;
        }
        else {
            std::cout << " Fb1a : " << timePerBlock.get(0, 1) / factor << "ms and occurence :" << occurencePerBlock.get(0, 1) << std::endl;
            std::cout << " Fb1b : " << timePerBlock.get(0, 2) / factor << "ms and occurence :" << occurencePerBlock.get(0, 2) << std::endl;
        }


        std::cout << " Fb2 : " << timePerBlock.get(0, 4) / factor << "ms and occurence :" << occurencePerBlock.get(0, 4) << std::endl;

        std::cout << " Fb3 : " << timePerBlock.get(0, 5) / factor << "ms and occurence :" << occurencePerBlock.get(0, 5) << std::endl;

        if (occurencePerBlock.get(0, 6) > 0) {
            std::cout << " Fb4 : " << timePerBlock.get(0, 6) / factor << "ms and occurence :" << occurencePerBlock.get(0, 6) << std::endl;

        }
    }
    else {
        std::cout << "pas de temps � afficher, ou alors il n'y a pas eut d'initialisation" << std::endl;
    }

    occurencePerBlock.saveCSV(fileName, mode);
    timePerBlock.saveCSV(fileName, mode);
}


template <unsigned int blockSize>
__global__ void calcW0GPU(float* W0, float* PQ, float* Cores, int N, int B) {
    __shared__ float shArr[blockSize];
    __shared__ float shArr2[blockSize];
    __shared__ bool mustCompute;

    int thIdx = threadIdx.x;
    int i = blockIdx.x;


    if (thIdx == 0) {
        mustCompute = false;
    }
    __syncthreads();
    float sum = 0;
    float sum2 = 0;
   
    for (int k = thIdx; k < N; k += blockSize) {
        if (Cores[k] == i) // c'est tr�s divergent, c'est nul
        {
            sum += PQ[k + 1];
            sum2 += PQ[k + 1 + N];
            mustCompute = true;
        } 
        /* ce n'est plus divergent, mais beaucoup plus d'acc�s m�moire...
         sum += Pinter[k] * (Cores[k] == i);
         sum2 += Qinter[k]* (Cores[k] == i);
        
        */
    }
    if (mustCompute) {
        shArr[thIdx] = sum;
        shArr2[thIdx] = sum2;
        __syncthreads();
        for (int size = blockSize / 2; size > 0; size /= 2) { //uniform
            if (thIdx < size) {
                shArr[thIdx] += shArr[thIdx + size];
                shArr2[thIdx] += shArr2[thIdx + size];
            }
            __syncthreads();
        }

        if (thIdx == 0) {
            W0[i] = shArr[0];
            W0[i + B] = shArr2[0];
        }
    }
    else {
        W0[i] = 0;
        W0[i + B] = 0;
    }
    
}



template <unsigned int blockSize>
__global__ void calcW0GPUD(double* W0D, double* PQD, float* Cores, int N, int B) {
    __shared__ double shArr[blockSize];
    __shared__ double shArr2[blockSize];
    __shared__ bool mustCompute;

    int thIdx = threadIdx.x;
    int i = blockIdx.x;

    if (thIdx == 0) {
        mustCompute = false;
    }
    __syncthreads();

    double sum = 0;
    double sum2 = 0;

    for (int k = thIdx; k < N-1; k += blockSize) {
        if ((int) Cores[k] == i) // c'est tr�s divergent, c'est nul
        {
            sum += PQD[k + 1];
            sum2 += PQD[k + 1 + N];
            mustCompute = true;
        }
        /* ce n'est plus divergent, mais beaucoup plus d'acc�s m�moire...
         sum += PQD[k] * (Cores[k] == i);
         sum2 += PQD[k + N]* (Cores[k] == i);

        */
    }
    if (mustCompute) {
        shArr[thIdx] = sum;
        shArr2[thIdx] = sum2;
        __syncthreads();
        for (int size = blockSize / 2; size > 0; size /= 2) { //uniform
            if (thIdx < size) {
                shArr[thIdx] += shArr[thIdx + size];
                shArr2[thIdx] += shArr2[thIdx + size];
            }
            __syncthreads();
        }

        if (thIdx == 0) {
            W0D[i] = shArr[0];
            W0D[i + B] = shArr2[0];
        }
    }
    else {
        W0D[i] = 0;
        W0D[i + B] = 0;
    }
}


template <unsigned int blockSize>
__global__ void calcW0GPUBis(float* W0, float* PQ, float* Cores, float* nAgentByBus, float* beginBus, int N, int B) {
    __shared__ float shArr[blockSize];
    __shared__ float shArr2[blockSize];
    __shared__ bool mustCompute;

    int thIdx = threadIdx.x;
    int i = blockIdx.x;
    int begin = beginBus[i];
    int end = begin + nAgentByBus[i];

    if (thIdx == 0) {
        mustCompute = nAgentByBus[i] > 0;
    }
    __syncthreads();
    if (mustCompute) { 
        float sum = 0;
        float sum2 = 0;
         for (int k = thIdx + begin; k < end; k += blockSize) {
             int indice = Cores[k];
             sum += PQ[indice];
             sum2 += PQ[indice + N];
         }
            
        shArr[thIdx] = sum;
        shArr2[thIdx] = sum2;
        __syncthreads();
        for (int size = blockSize / 2; size > 0; size /= 2) { //uniform
            if (thIdx < size) {
                shArr[thIdx] += shArr[thIdx + size];
                shArr2[thIdx] += shArr2[thIdx + size];
            }
            __syncthreads();
        }

        if (thIdx == 0) {
            W0[i] = shArr[0];
            W0[i + B] = shArr2[0];
        }
    }
}


template <unsigned int blockSize>
__global__ void calcW0GPUDBis(double* W0D, double* PQD, float* Cores, float* nAgentByBus, float* beginBus, int N, int B) {
    __shared__ double shArr[blockSize];
    __shared__ double shArr2[blockSize];
    __shared__ bool mustCompute;

    int thIdx = threadIdx.x;
    int i = blockIdx.x;
    int begin = beginBus[i];
    int end = begin + nAgentByBus[i];

    if (thIdx == 0) {
        mustCompute = nAgentByBus[i] > 0;
    }
    __syncthreads();
    if (mustCompute) {
        double sum = 0;
        double sum2 = 0;
        for (int k = thIdx + begin; k < end; k += blockSize) {
            int indice = Cores[k];
            sum += PQD[indice];
            sum2 += PQD[indice + N];
        }

        shArr[thIdx] = sum;
        shArr2[thIdx] = sum2;
        __syncthreads();
        for (int size = blockSize / 2; size > 0; size /= 2) { //uniform
            if (thIdx < size) {
                shArr[thIdx] += shArr[thIdx + size];
                shArr2[thIdx] += shArr2[thIdx + size];
            }
            __syncthreads();
        }

        if (thIdx == 0) {
            W0D[i] = shArr[0];
            W0D[i + B] = shArr2[0];
        }
    }
}


__global__ void initE(float* E, float theta0, float V0, int B) {


    int thIdx = threadIdx.x + blockDim.x * blockIdx.x;
    int size = gridDim.x * blockDim.x;
    for (int i = thIdx; i < B; i += size) {
        E[i] = theta0;
        E[i + B] = V0;
    }
}

__global__ void initED(double* ED, double theta0, double V0, int B) {


    int thIdx = threadIdx.x + blockDim.x * blockIdx.x;
    int size = gridDim.x * blockDim.x;
    for (int i = thIdx; i < B; i += size) {
        ED[i] = theta0;
        ED[i + B] = V0;
    }
}



template <unsigned int blockSize>
__global__ void calcWGPU(float* W, float* W0, float* Pinter, float* Qinter, float* CoresBusLin, float* nLines, int B) {
    __shared__ float shArr[blockSize];
    __shared__ float shArr2[blockSize];
    int thIdx = threadIdx.x;
    int i = blockIdx.x; // bus !!!

    int begin = CoresBusLin[i];
    int end = begin + nLines[i];

 
    float sum = 0;
    float sum2 = 0;
    if (i == 0) {
        if (thIdx == 0) {
            W[0] = W0[0];
            W[B] = W0[B];
        }
    }
    else {
        for (int i = begin + thIdx; i < end; i += blockSize) {
            sum += Pinter[i];
            sum2 += Qinter[i];
        }
        

        shArr[thIdx] = sum;
        shArr2[thIdx] = sum2;
        __syncthreads();
        for (int size = blockSize / 2; size > 0; size /= 2) { //uniform
            if (thIdx < size) {
                shArr[thIdx] += shArr[thIdx + size];
                shArr2[thIdx] += shArr2[thIdx + size];
            }
            
            __syncthreads();
        }
    
        if (thIdx == 0) {
            W[i] = shArr[0];
            W[i + B] = shArr2[0];
        }
    }
   
    
}

template <unsigned int blockSize>
__global__ void calcWGPU(float* W, float* Pinter, float* Qinter, float* CoresBusLin, float* nLines, int B) {
    __shared__ float shArr[blockSize];
    __shared__ float shArr2[blockSize];
    int thIdx = threadIdx.x;
    int i = blockIdx.x;
    int begin = CoresBusLin[i];
    int end = begin + nLines[i];


    float sum = 0;
    float sum2 = 0;
    for (int i = begin + thIdx; i < end; i += blockSize) {
        sum += Pinter[i];
        sum2 += Qinter[i];
    }


    shArr[thIdx] = sum;
    shArr2[thIdx] = sum2;
    __syncthreads();
    for (int size = blockSize / 2; size > 0; size /= 2) { //uniform
        if (thIdx < size) {
            shArr[thIdx] += shArr[thIdx + size];
            shArr2[thIdx] += shArr2[thIdx + size];
        }

        __syncthreads();
    }
    if (thIdx == 0) {
        W[i] = shArr[0];
        W[i + B] = shArr2[0];
    }
}

__global__ void calcWinter(float* Pinter,float*Qinter, float* E, float* Glin, float* Blin, float* CoresVoiLin, float* CoresBusLin, float* nLines, int B) {


    int index = threadIdx.x;
    int step = blockDim.x;
    int i = blockIdx.x;
    extern __shared__ float shE[];
    int begin = CoresBusLin[i];
    int end = begin + nLines[blockIdx.x];
    int B2 = 2 * B;

    for (int n = index; n < B2; n += step)
    {
        shE[n] = E[n];
    }
    __syncthreads();

    for (int l = begin + index; l < end; l += step) {
        int k = CoresVoiLin[l];
        float g = Glin[l];
        float b = Blin[l];
        float dt = shE[i] - shE[k];
        float cdt = cos(dt);
        float sdt = sin(dt);
        float v = shE[k + B] * shE[i + B];
       

        Pinter[l] = v * (g * cdt + b * sdt);
        Qinter[l] = v * (g * sdt - b * cdt);

    }

}

__global__ void calcJacGPU(float* Jac, float* W, float* E, float* Glin, float* Blin, float* CoresVoiLin, float* CoresBusLin, float* nLines, int B) {


    int index = threadIdx.x;
    int step = blockDim.x;
    int i = blockIdx.x;
    extern __shared__ float shE[];
    int begin = CoresBusLin[i];
    int end = begin + nLines[i];
    int B2 = 2 * B;

    for (int n = index; n < B2; n += step)
    {
        shE[n] = E[n];
    }
    __syncthreads();
    if (i != 0) {
        for (int l = begin + index; l < end; l += step) {
                int k = CoresVoiLin[l];
                float g = Glin[l];
                float b = Blin[l];
                float dt = shE[i] - shE[k];
                float cdt = cos(dt);
                float sdt = sin(dt);
                float vi = shE[i + B];
                float vk = shE[k + B];
                float p = W[i];
                float q = W[i + B];

                int i2 = i + B;
                int k2 = k + B;
        
      
                //
                Jac[i * B2 + k]   = (-q - b * vi * vi) * (i == k) + ( vi * vk * (g * sdt - b * cdt)) * (i != k);
                Jac[i * B2 + k2]  = (p / vi + g * vi)  * (i == k) + ( vi * (g * cdt + b * sdt)) * (i != k);
                Jac[i2 * B2 + k]  = (p - g * vi * vi)  * (i == k) + (-vi * vk * (g * cdt + b * sdt)) * (i != k);
                Jac[i2 * B2 + k2] = (q / vi - b * vi)  * (i == k) + ( vi * (g * sdt - b * cdt)) * (i != k);

        }
    }
    

}





template <unsigned int blockSize>
__global__ void calcWGPUD(double* W, double* W0, double* Pinter, double* Qinter, float* CoresBusLin, float* nLines, int B) {
    __shared__ double shArr[blockSize];
    __shared__ double shArr2[blockSize];
    int thIdx = threadIdx.x;
    int i = blockIdx.x;

    int begin = CoresBusLin[i];
    int end = begin + nLines[i];


    double sum = 0;
    double sum2 = 0;
    if (i == 0) {
        if (thIdx == 0) {
            W[0] = W0[0];
            W[B] = W0[B];
        }
    }
    else {
        for (int j = begin + thIdx; j < end; j += blockSize) {
            sum += Pinter[j];
            sum2 += Qinter[j];
        }


        shArr[thIdx] = sum;
        shArr2[thIdx] = sum2;
        __syncthreads();
        for (int size = blockSize / 2; size > 0; size /= 2) { //uniform
            if (thIdx < size) {
                shArr[thIdx] += shArr[thIdx + size];
                shArr2[thIdx] += shArr2[thIdx + size];
            }

            __syncthreads();
        }

        if (thIdx == 0) {
            W[i] = shArr[0];
            W[i + B] = shArr2[0];
        }
    }


}

template <unsigned int blockSize>
__global__ void calcWGPUD(double* W, double* Pinter, double* Qinter, float* CoresBusLin, float* nLines, int B) {
    __shared__ double shArr[blockSize];
    __shared__ double shArr2[blockSize];
    int thIdx = threadIdx.x;
    int i = blockIdx.x;
    int begin = CoresBusLin[i];
    int end = begin + nLines[i];


    double sum = 0;
    double sum2 = 0;
    for (int i = begin + thIdx; i < end; i += blockSize) {
        sum += Pinter[i];
        sum2 += Qinter[i];
    }


    shArr[thIdx] = sum;
    shArr2[thIdx] = sum2;
    __syncthreads();
    for (int size = blockSize / 2; size > 0; size /= 2) { //uniform
        if (thIdx < size) {
            shArr[thIdx] += shArr[thIdx + size];
            shArr2[thIdx] += shArr2[thIdx + size];
        }

        __syncthreads();
    }
    if (thIdx == 0) {
        W[i] = shArr[0];
        W[i + B] = shArr2[0];
    }
}

__global__ void calcWinterD(double* Pinter, double* Qinter, double* E, double* Glin, double* Blin, float* CoresVoiLin, float* CoresBusLin, float* nLines, int B) {


    int index = threadIdx.x;
    int step = blockDim.x;
    int i = blockIdx.x;
    extern __shared__ double shED[];
    int begin = CoresBusLin[i];
    int end = begin + nLines[blockIdx.x];
    int B2 = 2 * B;

    for (int n = index; n < B2; n += step)
    {
        shED[n] = E[n];
    }
    __syncthreads();

    for (int l = begin + index; l < end; l += step) {
        int k = CoresVoiLin[l];
        double g = Glin[l];
        double b = Blin[l];
        double dt = shED[i] - shED[k];
        double cdt = cos(dt);
        double sdt = sin(dt);
        double v = shED[k + B] * shED[i + B];


        Pinter[l] = v * (g * cdt + b * sdt);
        Qinter[l] = v * (g * sdt - b * cdt);

    }

}

__global__ void calcJacGPUD(double* Jac, double* W, double* E, double* Glin, double* Blin, float* CoresVoiLin, float* CoresBusLin, float* nLines, int B) {


    int index = threadIdx.x;
    int step = blockDim.x;
    int i = blockIdx.x;
    extern __shared__ double shED[];
    int begin = CoresBusLin[i];
    int end = begin + nLines[i];
    int B2 = 2 * B;
    __shared__ double p;
    __shared__ double q;
    if (index == 0) {
        p = W[i];
        q = W[i + B];
    }

    for (int n = index; n < B2; n += step)
    {
        shED[n] = E[n];
    }
    __syncthreads();
    if (i != 0) {
        for (int l = begin + index; l < end; l += step) {
            int k = CoresVoiLin[l];
            double g = Glin[l];
            double b = Blin[l];
            double dt = shED[i] - shED[k];
            double cdt = cos(dt);
            double sdt = sin(dt);
            double vi = shED[i + B];
            double vk = shED[k + B];


            int i2 = i + B;
            int k2 = k + B;


            //
            Jac[i * B2 + k] = (-q - b * vi * vi) * (i == k) + (vi * vk * (g * sdt - b * cdt)) * (i != k);
            Jac[i * B2 + k2] = (p / vi + g * vi) * (i == k) + (vi * (g * cdt + b * sdt)) * (i != k);
            Jac[i2 * B2 + k] = (p - g * vi * vi) * (i == k) + (-vi * vk * (g * cdt + b * sdt)) * (i != k);
            Jac[i2 * B2 + k2] = (q / vi - b * vi) * (i == k) + (vi * (g * sdt - b * cdt)) * (i != k);

        }
    }


}