#include "hip/hip_runtime.h"
#include "../head/ADMMGPUConstCons2.cuh"

ADMMGPUConstCons2::ADMMGPUConstCons2() : MethodP2PGPU()
{
#if DEBUG_CONSTRUCTOR
	std::cout << "Constructeur ADMMGPUConstCons2" << std::endl;
#endif // DEBUG_CONSTRUCTOR
	_name = NAME;
}

ADMMGPUConstCons2::ADMMGPUConstCons2(float rho) : MethodP2PGPU()
{
#if DEBUG_CONSTRUCTOR
	std::cout << "Constructeur ADMMGPUConstCons2 defaut" << std::endl;
#endif // DEBUG_CONSTRUCTOR
	_name = NAME;
	_rho = rho;
}

ADMMGPUConstCons2::~ADMMGPUConstCons2()
{
	
}

void ADMMGPUConstCons2::setParam(float rho)
{
	_rho = rho;
}

void ADMMGPUConstCons2::setTau(float tau)
{
	if (tau < 1) {
		throw std::invalid_argument("tau must be greater than 1");
	}
	_tau = tau;
}

void ADMMGPUConstCons2::init(const Simparam& sim, const StudyCase& cas)
{
	// intitilisation des matrixs et variables 
	_rhog = sim.getRho();
	_rho1 = sim.getRho1();
	
	//std::cout << "rho initial " << _rhog << std::endl;
	_nAgent = sim.getNAgent();
	
	_rhol = _rho;
	if (_rho == 0) {
		_rhol = _rhog;
	}
	const int iterG = sim.getIterG();
	const int stepG = sim.getStepG();
	kmax   = sim.getIterIntern();
	epsOPF = sim.getEpsIntern();
	float epsG = sim.getEpsG();
	float epsGC = sim.getEpsGC();
	_ratioEps = epsG / epsGC;
	
	nVoisinCPU = cas.getNvoi();
	nVoisin = MatrixGPU(nVoisinCPU, 1);

	int nVoisinMax = nVoisin.max2();
	if (_blockSize * NMAXPEERPERTRHREAD < nVoisinMax) {
		std::cout << _blockSize << " " << NMAXPEERPERTRHREAD << " " << nVoisinMax << std::endl;
		throw std::invalid_argument("For this Method, an agent must not have more than 5120 peers");
	}

	_nLine = cas.getNLine();
	L2 = 2 * _nLine;
	_Msize = _nAgent + L2 + 1;
	_Asize = L2 * _nAgent;
	std::cout << _nAgent << " " << _nLine << " " << _Msize << std::endl;
	
	_nBus = cas.getNBus();

	_nTrade = nVoisin.sum();
	_numBlocksN = ceil((_nAgent + _blockSize - 1) / _blockSize);
	_numBlocksM = ceil((_nTrade + _blockSize - 1) / _blockSize);
	_numBlocksL = ceil((_nLine + _blockSize - 1) / _blockSize);
	_numBlocksNL = ceil((_nAgent * _nLine + _blockSize - 1) / _blockSize);
	_at1 = _rhog; // represente en fait 2*a
	_at2 = _rhol;

	resF = MatrixCPU(3, (iterG / stepG) + 1);
	resX = MatrixCPU(4, (iterG / stepG) + 1);

	MatrixCPU BETA(cas.getBeta());
	MatrixGPU Ub(cas.getUb());
	MatrixGPU Lb(cas.getLb());
	LAMBDA = sim.getLambda();
	trade = sim.getTrade();
	
	//std::cout << "mise sous forme lin�aire" << std::endl;
	// Rem : si matrice d�j� existante, elles sont d�j� sur GPU donc bug pour les get

	CoresMatLin = MatrixGPU(_nAgent, _nAgent, -1);
	CoresLinAgent = MatrixGPU(_nTrade, 1);
	CoresAgentLin = MatrixGPU(_nAgent + 1, 1);
	CoresLinVoisin = MatrixGPU(_nTrade, 1);
	CoresLinTrans = MatrixGPU(_nTrade, 1);
	
	Tlocal_pre = MatrixGPU(_nTrade, 1);
	tradeLin = MatrixGPU(_nTrade, 1);
	LAMBDALin = MatrixGPU(_nTrade, 1);

	matLb = MatrixGPU(_nTrade, 1);
	matUb = MatrixGPU(_nTrade, 1);
	Ct = MatrixGPU(_nTrade, 1);

	int indice = 0;

	for (int idAgent = 0; idAgent < _nAgent; idAgent++) {
		MatrixCPU omega(cas.getVoisin(idAgent));
		int Nvoisinmax = nVoisinCPU.get(idAgent, 0);
		for (int voisin = 0; voisin < Nvoisinmax; voisin++) {
			int idVoisin = omega.get(voisin, 0);
			if(Lb.getNCol()==1){
				matLb.set(indice, 0, Lb.get(idAgent, 0));
				matUb.set(indice, 0, Ub.get(idAgent, 0));
			} else {
				matLb.set(indice, 0, Lb.get(idAgent, idVoisin));
				matUb.set(indice, 0, Ub.get(idAgent, idVoisin));
			}
			Ct.set(indice, 0, BETA.get(idAgent, idVoisin));
			tradeLin.set(indice, 0, trade.get(idAgent, idVoisin));
			Tlocal_pre.set(indice, 0, trade.get(idAgent, idVoisin));
			LAMBDALin.set(indice, 0, LAMBDA.get(idAgent, idVoisin));
			CoresLinAgent.set(indice, 0, idAgent);
			CoresLinVoisin.set(indice, 0, idVoisin);
			CoresMatLin.set(idAgent, idVoisin, indice);
			indice = indice + 1;
		}
		CoresAgentLin.set(idAgent + 1, 0, indice);
	}
	for (int lin = 0; lin < _nTrade; lin++) {
		int i = CoresLinAgent.get(lin, 0);
		int j = CoresLinVoisin.get(lin, 0);
		int k = CoresMatLin.get(j, i);
		CoresLinTrans.set(lin, 0, k);
	}


	// transfert des mises lineaire
	matUb.transferGPU();
	matLb.transferGPU();
	Ct.transferGPU();

	Tlocal_pre.transferGPU();
	tradeLin.transferGPU();
	LAMBDALin.transferGPU();

	CoresAgentLin.transferGPU();
	CoresLinAgent.transferGPU();
	CoresLinVoisin.transferGPU();
	CoresMatLin.transferGPU();
	CoresLinTrans.transferGPU();
	
	
	
	//std::cout << "donnees sur GPU pour le grid" << std::endl;
	if (_nLine) {
		H = MatrixGPU(_nAgent, _nAgent, 0);
		H.setEyes(_rho1);
		q = MatrixGPU(_nAgent, 1, 0); // 0.5x^THx + q^T*x

		c = MatrixGPU(L2 + 1, 1, 0); // contrainte Ax+b>0 ou = 0 pour egalit�
		Ai = MatrixGPU(L2 + 1, _nAgent, 0);
		MatrixGPU ones(1, _nAgent, 1);
		MatrixGPU temp(cas.getPowerSensi());
		Ai.setBloc(0, _nLine, 0, _nAgent, &temp, -1);
		Ai.setBloc(_nLine, L2, 0, _nAgent, &temp);
		Ai.setBloc(L2, L2 + 1, 0, _nAgent, &ones);
		bi = MatrixGPU(L2 + 1, 1, 0);
		lLimit = MatrixGPU(cas.getLineLimit());
		bi.setBloc(0, _nLine, 0, 1, &lLimit);
		bi.setBloc(_nLine, L2, 0, 1, &lLimit);

		M = MatrixGPU(_Msize, _Msize, 0); // M*pas = R
		Minv = MatrixGPU(_Msize, _Msize, 0); // M*pas = R
		pas = MatrixGPU(_Msize, 1, 0);
		R = MatrixGPU(_Msize, 1, 0);


		ZA = MatrixGPU(L2 + 1, _nAgent, 0); // M = (H -Atrans ZA W)
		Z = MatrixGPU(L2 + 1, L2 + 1, 0);
		Zvect = MatrixGPU(L2 + 1, 1, 0);
		W = MatrixGPU(L2 + 1, L2 + 1, 0);
		Wvect = MatrixGPU(L2 + 1, 1, 0);
		Atrans = MatrixGPU(_nAgent, L2 + 1, 0);
		Atrans.setTrans(&Ai);

		M.setBloc(0, _nAgent, 0, _nAgent, &H);
		M.setBloc(0, _nAgent, _nAgent, _Msize, &Atrans, -1);

			

		Rx1 = MatrixGPU(_nAgent, 1, 0); // Hx+q
		Rx2 = MatrixGPU(_nAgent, 1, 0); // -Ai^T*U

		Ru = MatrixGPU(L2 + 1, 1, 0); // Ru = W*(U-PI)

		U = MatrixGPU(L2 + 1, 1, 0);
		PI = MatrixGPU(L2 + 1, 1, 0);			

	}
	Pso = MatrixGPU(_nAgent, 1, 0, 1); // = Pn ? risque de non respect des contraintes
	
	etaP = MatrixGPU(_nAgent, 1, 0, 1); 

	//std::cout << "autres donn�e sur GPU" << std::endl;
	tempNN = MatrixGPU(_nTrade, 1, 0, 1);
	tempN1 = MatrixGPU(_nAgent, 1, 0, 1); // plut�t que de re-allouer de la m�moire � chaque utilisation
	tempL1 = MatrixGPU(_nLine, 1, 0, 1);
	tempL2 = MatrixGPU(_nLine, 1, 0, 1);
	//MatrixGPU temp1N(1, _nAgent, 0, 1);

	Tlocal = MatrixGPU(_nTrade, 1, 0, 1);
	P = MatrixGPU(_nAgent, 1, 0, 1); // moyenne des trades
	Pn = MatrixGPU(sim.getPn(), 1); // somme des trades


	a = MatrixGPU(cas.geta(), 1);
	b = MatrixGPU(cas.getb(), 1);
	Ap2 = a;
	Ap1 = nVoisin;
	Ap3 = nVoisin;
	Ap123 = MatrixGPU(_nAgent, 1, 0, 1);
	Bp3 = MatrixGPU(_nAgent, 1, 0, 1); // 1/Mn * (Pso + P)/2 - eta/rho1

	Bt1 = MatrixGPU(_nTrade, 1, 0, 1);
	Cp = b;

	Pmin = MatrixGPU(cas.getPmin(), 1);
	Pmax = MatrixGPU(cas.getPmax(), 1);
	MU = MatrixGPU(sim.getMU(), 1); // facteur reduit i.e lambda_l/_rho
	Tmoy = MatrixGPU(sim.getPn(), 1);

	tempNN.preallocateReduction();
	Tlocal.preallocateReduction();
	tempL1.preallocateReduction();
	

	P.preallocateReduction();
	Pso.preallocateReduction();


	Pmin.divideT(&nVoisin);
	Pmax.divideT(&nVoisin);
	Ap1.multiply(_rhol);
	Ap3.multiplyT(&nVoisin);
	Ap3.multiply(_rho1);
	Cp.multiplyT(&nVoisin);
	Tmoy.divideT(&nVoisin);

	Ap2.multiplyT(&nVoisin);
	Ap2.multiplyT(&nVoisin);
	Ap123.add(&Ap1, &Ap2);
	Ap123.add(&Ap3);
	
	
	
	updateGlobalProbGPU();

	//Hosqp.display();
	
	//std::cout << " end init " << std::endl;
}



void ADMMGPUConstCons2::updateP0(const StudyCase& cas)
{
	_id = _id + 1;
#ifdef INSTRUMENTATION
	hipDeviceSynchronize();
	std::chrono::high_resolution_clock::time_point t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION

	matLb.transferCPU();
	
	Pmin = MatrixGPU(cas.getPmin());
	Pmax = MatrixGPU(cas.getPmax());


	MatrixGPU Lb(cas.getLb());

	b = cas.getb();
	Cp = cas.getb();
	int indice = 0;

	for (int idAgent = 0; idAgent < _nAgent; idAgent++) {
		int Nvoisinmax = nVoisinCPU.get(idAgent, 0);
		for (int voisin = 0; voisin < Nvoisinmax; voisin++) {
			matLb.set(indice, 0, Lb.get(idAgent, 0));
			indice = indice + 1;
		}
	}

	
	matLb.transferGPU();

	Pmin.divideT(&nVoisin);
	Pmax.divideT(&nVoisin);
	Cp.multiplyT(&nVoisin);
#ifdef INSTRUMENTATION
	hipDeviceSynchronize();
	std::chrono::high_resolution_clock::time_point t2 = std::chrono::high_resolution_clock::now();
	timePerBlock.increment(0, 10, (float) std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
	occurencePerBlock.increment(0, 10, 1);
#endif // INSTRUMENTATION

	//std::cout << "fin update temps : " << (float)(clock() - t) / CLOCKS_PER_SEC << std::endl;
}

void ADMMGPUConstCons2::solveOPF()
{
	
	// update q
	MatrixGPU diffP(Pso);
	q.set(&etaP);
	diffP.add(&Pn);
	diffP.multiply(-_rho1/2);
	q.add(&diffP);
	
	//init
	int k = 0;
	float err = 2 * epsOPF;
	mu = 10;
	MatrixGPU tempL21(L2 + 1, 1, 0);
	float valMin = 0.0000001;
	//boucle
	while (k<kmax && err>epsOPF) {
	// update c
		
		c.linearOperation(&Ai, &Pso, &bi);
		
	// update PI
		for (int l = 0; l < L2; l++) {
			if (c.get(l, 0) < valMin) {
				PI.set(l, 0, mu / valMin); // eviter division par O
			}
			else {
				PI.set(l, 0, mu / c.get(l, 0)); // eviter division par O
			}
		}
		PI.set(L2, 0, -c.get(L2, 0) / mu);
	
	// update M
		// update Zvect
		
		Zvect.set(&U); 
		
		Zvect.set(L2, 0, 1); // egalite
		// update Z
		Z.setEyes(&Zvect);
		// update ZA
		ZA.multiplyMat(&Z, &Ai);
		// update W
		Wvect.set(&c);
		Wvect.set(L2, 0, mu);
		W.setEyes(&Wvect);

		M.setBloc(_nAgent, _Msize, 0, _nAgent, &ZA);
		M.setBloc(_nAgent, _Msize, _nAgent, _Msize, &W);
		try
		{
			Minv.invertGaussJordan(&M);
		}
		catch (const std::exception& e)
		{
			std::cout << e.what() << std::endl;
			
			std::cout << "k = " << k << " err= " << err << " alpha = " << alpha << " mu=" << mu << std::endl;
			c.display();
			Pn.display();
			Pso.display();
			std::cout << "---------------------------------" << std::endl;
			Pso.set(&Pn);
			return;
		}
		
	
	//update R
		// Rx
		Rx1.linearOperation(&H, &Pso, &q);
		Rx2.multiply(&Atrans, &U);
		Rx2.subtract(&Rx1);
		// Ru
		tempL21.subtract(&PI, &U);
		Ru.multiply(&W, &tempL21);

		R.setBloc(0, _nAgent, 0, 1, &Rx2);
		R.setBloc(_nAgent, _Msize, 0, 1, &Ru);
		//update pas
		pas.multiply(&Minv, &R);
		// find alpha
		findalpha();
		// update P, U
		pas.multiply(alpha);
		for (int n = 0; n < _nAgent; n++) {
			Pso.set(n, 0, Pso.get(n, 0) + pas.get(n, 0));
		}
		for (int l = 0; l < L2 + 1; l++) {
			U.set(l, 0, U.get(l, 0) + pas.get(_nAgent + l, 0));
		}

		// update mu
		mu *= 0.8;
		mu = MYMAX(mu, valMin);

		err = R.distance2();
		k++;
	}
	//std::cout << k << " " << err << std::endl;


}


void ADMMGPUConstCons2::findalpha()
{
	//version sur CPU
	alpha = 1;
	// U = U + alpha * pas >0
	for (int l = 0; l < L2; l++) {
		float div = pas.get(l + _nAgent, 0);
		if (div<0) {
			float newAlpha = -U.get(l, 0) / div;
			alpha = alpha < newAlpha ? alpha : newAlpha;
		}
	}
	// c > 0
	

	for (int l = 0; l < L2; l++) {
		float sum = 0;
		for (int p = 0; p < _nAgent; p++) {
			sum += Ai.get(l, p) * pas.get(p, 0);
		}
		if (sum < 0) {
			float newAlpha = -c.get(l, 0) / sum;
			alpha = alpha < newAlpha ? alpha : newAlpha;
		}
	}

	alpha *= 0.9;


}


void ADMMGPUConstCons2::solve(Simparam* result, const Simparam& sim, const StudyCase& cas)
{
#ifdef DEBUG_SOLVE
	cas.display();
	sim.display(1);
#endif // DEBUG_SOLVE
	
	
	clock_t tall = clock();
#ifdef INSTRUMENTATION
	std::chrono::high_resolution_clock::time_point t1;
	std::chrono::high_resolution_clock::time_point t2;
#endif // INSTRUMENTATION

	if (_id == 0) {
#ifdef INSTRUMENTATION
		hipDeviceSynchronize();
		t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION
		init(sim, cas);
#ifdef INSTRUMENTATION
		hipDeviceSynchronize();
		t2 = std::chrono::high_resolution_clock::now();
		timePerBlock.increment(0, 0, (float) std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
		occurencePerBlock.increment(0, 0, 1);
#endif // INSTRUMENTATION
	}
	//std::cout << _numBlocks2 << " " <<  _blockSize << std::endl;
	
	_at1 = _rhog; // represente en fait 2*a
	
	float epsG = sim.getEpsG();
	float epsL = sim.getEpsL();
	const int stepL = sim.getStepL();
	const int stepG = sim.getStepG();
	const int iterG = sim.getIterG();
	const int iterL = sim.getIterL();
	

	float resG = 2 * epsG;
	float epsL2 = epsL * epsL;
	int iterGlobal = 0;
	
	
	//std::cout << iterG << " " << iterL << " " << epsL << " " << epsG << std::endl;
	while ((iterGlobal < iterG) && (resG > epsG)) {
#ifdef INSTRUMENTATION
		hipDeviceSynchronize();
		t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION

		updateLocalProbGPU(epsL2, iterL);
#ifdef INSTRUMENTATION
		hipDeviceSynchronize();
		t2 = std::chrono::high_resolution_clock::now();
		timePerBlock.increment(0, 1, (float) std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
#endif // INSTRUMENTATION

		tradeLin.swap(&Tlocal); // echange juste les pointeurs	
		//std::cout << "-";
		
		updateGlobalProbGPU();
		
		if (!(iterGlobal % stepG)) {
#ifdef INSTRUMENTATION
			hipDeviceSynchronize();
			t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION
			resG = updateRes(iterGlobal / stepG);
#ifdef INSTRUMENTATION
			hipDeviceSynchronize();
			t2 = std::chrono::high_resolution_clock::now();
			timePerBlock.increment(0, 8, (float) std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
#endif // INSTRUMENTATION
		}
		//std::cout << iterGlobal << " " << iterLocal << " " << resL << " " << resF.get(0, iterGlobal / stepG) << " " << resF.get(1, iterGlobal / stepG) << std::endl;
		iterGlobal++;
	}
#ifdef INSTRUMENTATION
	occurencePerBlock.increment(0, 1, iterGlobal);
	occurencePerBlock.increment(0, 5, iterGlobal);
	occurencePerBlock.increment(0, 6, iterGlobal);
	occurencePerBlock.increment(0, 7, iterGlobal);
	occurencePerBlock.increment(0, 8, iterGlobal / stepG);

	hipDeviceSynchronize();
	t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION
	
	

	float fc = calcFc();
	std::cout << "valeur finale des contraintes de l'opf : " << std::endl;
	c.display();
	//std::cout << iterGlobal << " " << iterLocal << " " << resL << " " << resG << std::endl;
	MatrixCPU tradeLinCPU;
	tradeLin.toMatCPU(tradeLinCPU);
	MatrixCPU LAMBDALinCPU;
	LAMBDALin.toMatCPU(LAMBDALinCPU);
	MatrixCPU PnCPU;
	Pn.toMatCPU(PnCPU);
	MatrixCPU MUCPU;
	MU.toMatCPU(MUCPU);
	

	int indice = 0;
	for (int idAgent = 0;idAgent < _nAgent; idAgent++) {
		MatrixCPU omega(cas.getVoisin(idAgent));
		int Nvoisinmax = nVoisinCPU.get(idAgent, 0);
		for (int voisin = 0; voisin < Nvoisinmax; voisin++) {
			int idVoisin = omega.get(voisin, 0);
			trade.set(idAgent, idVoisin, tradeLinCPU.get(indice, 0));
			LAMBDA.set(idAgent, idVoisin, LAMBDALinCPU.get(indice, 0));
			indice = indice + 1;
		}
	}
	result->setResF(&resF);
	result->setLAMBDA(&LAMBDA);
	result->setTrade(&trade);
	
	result->setIter(iterGlobal);
	result->setPn(&PnCPU);
	result->setFc(fc);
	result->setMU(&MUCPU);
	result->setRho(_rhog);
#ifdef INSTRUMENTATION
	hipDeviceSynchronize();
	t2 = std::chrono::high_resolution_clock::now();
	timePerBlock.increment(0, 9, (float) std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
	occurencePerBlock.increment(0, 9, 1);
	result->setTimeBloc(&timePerBlock, &occurencePerBlock);
#endif // INSTRUMENTATION

	tall = clock() - tall;
	result->setTime((float)tall / CLOCKS_PER_SEC);
}

void ADMMGPUConstCons2::updateLocalProbGPU(float epsL, int nIterL) {
	int numBlocks = _nAgent;
	switch (_blockSize) {
	case 512:
		updateTradePGPUSharedResidualCons<512> << <numBlocks, _blockSize >> > (Tlocal._matrixGPU, Tlocal_pre._matrixGPU, Tmoy._matrixGPU, P._matrixGPU, MU._matrixGPU, nVoisin._matrixGPU, _at1, _at2, Bt1._matrixGPU, Ct._matrixGPU,
			matLb._matrixGPU, matUb._matrixGPU, Ap1._matrixGPU, Ap3._matrixGPU, Ap123._matrixGPU, Bp3._matrixGPU, Cp._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, CoresAgentLin._matrixGPU, epsL, nIterL);
		break;
	case 256:
		updateTradePGPUSharedResidualCons<256> << <numBlocks, _blockSize >> > (Tlocal._matrixGPU, Tlocal_pre._matrixGPU, Tmoy._matrixGPU, P._matrixGPU, MU._matrixGPU, nVoisin._matrixGPU, _at1, _at2, Bt1._matrixGPU, Ct._matrixGPU,
			matLb._matrixGPU, matUb._matrixGPU, Ap1._matrixGPU, Ap3._matrixGPU, Ap123._matrixGPU, Bp3._matrixGPU, Cp._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, CoresAgentLin._matrixGPU, epsL, nIterL);
		break;
	case 128:
		updateTradePGPUSharedResidualCons<128> << <numBlocks, _blockSize >> > (Tlocal._matrixGPU, Tlocal_pre._matrixGPU, Tmoy._matrixGPU, P._matrixGPU, MU._matrixGPU, nVoisin._matrixGPU, _at1, _at2, Bt1._matrixGPU, Ct._matrixGPU,
			matLb._matrixGPU, matUb._matrixGPU, Ap1._matrixGPU, Ap3._matrixGPU, Ap123._matrixGPU, Bp3._matrixGPU, Cp._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, CoresAgentLin._matrixGPU, epsL, nIterL);
		break;
	case 64:
		updateTradePGPUSharedResidualCons< 64> << <numBlocks, _blockSize >> > (Tlocal._matrixGPU, Tlocal_pre._matrixGPU, Tmoy._matrixGPU, P._matrixGPU, MU._matrixGPU, nVoisin._matrixGPU, _at1, _at2, Bt1._matrixGPU, Ct._matrixGPU,
			matLb._matrixGPU, matUb._matrixGPU, Ap1._matrixGPU, Ap3._matrixGPU, Ap123._matrixGPU, Bp3._matrixGPU, Cp._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, CoresAgentLin._matrixGPU, epsL, nIterL);
		break;
	case 32:
		updateTradePGPUSharedResidualCons< 32> << <numBlocks, _blockSize >> > (Tlocal._matrixGPU, Tlocal_pre._matrixGPU, Tmoy._matrixGPU, P._matrixGPU, MU._matrixGPU, nVoisin._matrixGPU, _at1, _at2, Bt1._matrixGPU, Ct._matrixGPU,
			matLb._matrixGPU, matUb._matrixGPU, Ap1._matrixGPU, Ap3._matrixGPU, Ap123._matrixGPU, Bp3._matrixGPU, Cp._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, CoresAgentLin._matrixGPU, epsL, nIterL);
		break;
	case 16:
		updateTradePGPUSharedResidualCons< 16> << <numBlocks, _blockSize >> > (Tlocal._matrixGPU, Tlocal_pre._matrixGPU, Tmoy._matrixGPU, P._matrixGPU, MU._matrixGPU, nVoisin._matrixGPU, _at1, _at2, Bt1._matrixGPU, Ct._matrixGPU,
			matLb._matrixGPU, matUb._matrixGPU, Ap1._matrixGPU, Ap3._matrixGPU, Ap123._matrixGPU, Bp3._matrixGPU, Cp._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, CoresAgentLin._matrixGPU, epsL, nIterL);
		break;
	case  8:
		updateTradePGPUSharedResidualCons<  8> << <numBlocks, _blockSize >> > (Tlocal._matrixGPU, Tlocal_pre._matrixGPU, Tmoy._matrixGPU, P._matrixGPU, MU._matrixGPU, nVoisin._matrixGPU, _at1, _at2, Bt1._matrixGPU, Ct._matrixGPU,
			matLb._matrixGPU, matUb._matrixGPU, Ap1._matrixGPU, Ap3._matrixGPU, Ap123._matrixGPU, Bp3._matrixGPU, Cp._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, CoresAgentLin._matrixGPU, epsL, nIterL);
		break;
	case  4:
		updateTradePGPUSharedResidualCons<  4> << <numBlocks, _blockSize >> > (Tlocal._matrixGPU, Tlocal_pre._matrixGPU, Tmoy._matrixGPU, P._matrixGPU, MU._matrixGPU, nVoisin._matrixGPU, _at1, _at2, Bt1._matrixGPU, Ct._matrixGPU,
			matLb._matrixGPU, matUb._matrixGPU, Ap1._matrixGPU, Ap3._matrixGPU, Ap123._matrixGPU, Bp3._matrixGPU, Cp._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, CoresAgentLin._matrixGPU, epsL, nIterL);
		break;
	case  2:
		updateTradePGPUSharedResidualCons<  2> << <numBlocks, _blockSize >> > (Tlocal._matrixGPU, Tlocal_pre._matrixGPU, Tmoy._matrixGPU, P._matrixGPU, MU._matrixGPU, nVoisin._matrixGPU, _at1, _at2, Bt1._matrixGPU, Ct._matrixGPU,
			matLb._matrixGPU, matUb._matrixGPU, Ap1._matrixGPU, Ap3._matrixGPU, Ap123._matrixGPU, Bp3._matrixGPU, Cp._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, CoresAgentLin._matrixGPU, epsL, nIterL);
		break;
	case  1:
		updateTradePGPUSharedResidualCons<  1> << <numBlocks, _blockSize >> > (Tlocal._matrixGPU, Tlocal_pre._matrixGPU, Tmoy._matrixGPU, P._matrixGPU, MU._matrixGPU, nVoisin._matrixGPU, _at1, _at2, Bt1._matrixGPU, Ct._matrixGPU,
			matLb._matrixGPU, matUb._matrixGPU, Ap1._matrixGPU, Ap3._matrixGPU, Ap123._matrixGPU, Bp3._matrixGPU, Cp._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, CoresAgentLin._matrixGPU, epsL, nIterL);
		break;
	}
	//hipStreamSynchronize(streamCalculation);
}



void ADMMGPUConstCons2::updateGlobalProbGPU()
{
	//Rem : tout calcul qui est de taille N ou M peut �tre fait par les agents
		// Si le calcul est de taile L, soit c'est calcul� par un/des superviseurs, soit tous les agents le calcul (un peu absurde)

#ifdef INSTRUMENTATION
// FB 3a
	hipDeviceSynchronize();
	std::chrono::high_resolution_clock::time_point t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION


	updatePnGPU << <_numBlocksN, _blockSize >> > (Pn._matrixGPU, Tmoy._matrixGPU, nVoisin._matrixGPU, _nAgent);
	
#ifdef INSTRUMENTATION
	hipDeviceSynchronize();
	std::chrono::high_resolution_clock::time_point t2 = std::chrono::high_resolution_clock::now();

	timePerBlock.increment(0, 5, (float) std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());

	// FB 3b
	hipDeviceSynchronize();
	t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION	

	// Resolution de l'OPF
	if (_nLine) {
		
		Pso.transferCPU();
		Pn.transferCPU();
		etaP.transferCPU();
		/*std::cout << " Pn :" << std::endl;
		Pn.display();
		std::cout << " etaP :" << std::endl;
		etaP.display();*/

		solveOPF();
		/*std::cout << " Pso :" << std::endl;
		Pso.display();*/
		Pn.transferGPU();
		etaP.transferGPU();
		Pso.transferGPU();
		
	}
	else {
		Pso = Pn;
	}
	
	
	

#ifdef INSTRUMENTATION
	hipDeviceSynchronize();
	t2 = std::chrono::high_resolution_clock::now();
	timePerBlock.increment(0, 6, (float) std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());

	// FB 3c
	hipDeviceSynchronize();
	t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION

	// update Bp3
	updateEtaPBp3 << <_numBlocksN, _blockSize >> > (Bp3._matrixGPU, etaP._matrixGPU, nVoisin._matrixGPU, Pso._matrixGPU, Pn._matrixGPU, _rho1, _nAgent);
	updateLAMBDABt1GPU << <_numBlocksM, _blockSize >> > (Bt1._matrixGPU, LAMBDALin._matrixGPU, tradeLin._matrixGPU, _rhog, CoresLinTrans._matrixGPU, _nTrade);
	

#ifdef INSTRUMENTATION
	hipDeviceSynchronize();
	t2 = std::chrono::high_resolution_clock::now();
	timePerBlock.increment(0, 7, (float) std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
#endif // INSTRUMENTATION
}



float ADMMGPUConstCons2::updateResEndo(int iter)
{

	float resS = Tlocal.max2(&tradeLin);

	updateDiffGPU <<<_numBlocksM, _blockSize >>> (tempNN._matrixGPU, Tlocal._matrixGPU, CoresLinTrans._matrixGPU, _nAgent);
	float resR = tempNN.max2();

	float resXf = _ratioEps * Pso.max2(&Pn);
	resF.set(0, iter, resR);
	resF.set(1, iter, resS);
	resF.set(2, iter, resXf);
	return MYMAX(MYMAX(resXf, resS), resR);

}





