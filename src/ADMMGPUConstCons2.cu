#include "hip/hip_runtime.h"
#include "../head/ADMMGPUConstCons2.cuh"

ADMMGPUConstCons2::ADMMGPUConstCons2() : MethodP2PGPU()
{
#if DEBUG_CONSTRUCTOR
	std::cout << "Constructeur ADMMGPUConstCons2" << std::endl;
#endif // DEBUG_CONSTRUCTOR
	_name = NAME;
	timePerBlock = MatrixCPU(1, 9, 0); // Fb0, Fb1 , Fb2, Fb3, Fb5, Fb6 Fb0'
	// si les sous ensemble ne sont pas accessible, tout est dans le premier.
	occurencePerBlock = MatrixCPU(1, 9, 0); //nb de fois utilis� pendant la simu

}

ADMMGPUConstCons2::ADMMGPUConstCons2(float rho) : MethodP2PGPU()
{
#if DEBUG_CONSTRUCTOR
	std::cout << "Constructeur ADMMGPUConstCons2 defaut" << std::endl;
#endif // DEBUG_CONSTRUCTOR
	timePerBlock = MatrixCPU(1, 9, 0); // Fb0, Fb1 , Fb2, Fb3, Fb5, Fb6 Fb0'
	// si les sous ensemble ne sont pas accessible, tout est dans le premier.
	occurencePerBlock = MatrixCPU(1, 9, 0); //nb de fois utilis� pendant la simu

	_name = NAME;
	_rho = rho;
}

ADMMGPUConstCons2::~ADMMGPUConstCons2()
{
	
}

void ADMMGPUConstCons2::setParam(float rho)
{
	_rho = rho;
}

void ADMMGPUConstCons2::setTau(float tau)
{
	if (tau < 1) {
		throw std::invalid_argument("tau must be greater than 1");
	}
	_tau = tau;
}

void ADMMGPUConstCons2::init(const Simparam& sim, const StudyCase& cas)
{
	// intitilisation des matrixs et variables 
	isAC = false;
	initSize(cas);
	initSimParam(sim);
		
	int nVoisinMax = nVoisin.max2();
	if (_blockSize * NMAXPEERPERTRHREAD < nVoisinMax) {
		std::cout << _blockSize << " " << NMAXPEERPERTRHREAD << " " << nVoisinMax << std::endl;
		throw std::invalid_argument("For this Method, an agent must not have more than 5120 peers");
	}

	L2 = 2 * _nLine;
	_Msize = _nAgent + L2 + 1;
	_Asize = L2 * _nAgent;
	//std::cout << _nAgent << " " << _nLine << " " << _Msize << std::endl;
	initCaseParam(sim, cas);
	//std::cout << "mise sous forme lineaire" << std::endl;
	initLinForm(cas);
		
	//std::cout << "donnees sur GPU pour le grid" << std::endl;
	if (_nLine) {
		H = MatrixGPU(_nAgent, _nAgent, 0);
		H.setEyes(_rho1);
		q = MatrixGPU(_nAgent, 1, 0); // 0.5x^THx + q^T*x

		c = MatrixGPU(L2 + 1, 1, 0); // contrainte Ax+b>0 ou = 0 pour egalit�
		Ai = MatrixGPU(L2 + 1, _nAgent, 0);
		MatrixGPU ones(1, _nAgent, 1);
		MatrixGPU temp(cas.getPowerSensi());
		Ai.setBloc(0, _nLine, 0, _nAgent, &temp, -1);
		Ai.setBloc(_nLine, L2, 0, _nAgent, &temp);
		Ai.setBloc(L2, L2 + 1, 0, _nAgent, &ones);
		bi = MatrixGPU(L2 + 1, 1, 0);
		lLimit = MatrixGPU(cas.getLineLimit());
		bi.setBloc(0, _nLine, 0, 1, &lLimit);
		bi.setBloc(_nLine, L2, 0, 1, &lLimit);

		M = MatrixGPU(_Msize, _Msize, 0); // M*pas = R
		Minv = MatrixGPU(_Msize, _Msize, 0); // M*pas = R
		pas = MatrixGPU(_Msize, 1, 0);
		R = MatrixGPU(_Msize, 1, 0);


		ZA = MatrixGPU(L2 + 1, _nAgent, 0); // M = (H -Atrans ZA W)
		Z = MatrixGPU(L2 + 1, L2 + 1, 0);
		Zvect = MatrixGPU(L2 + 1, 1, 0);
		W = MatrixGPU(L2 + 1, L2 + 1, 0);
		Wvect = MatrixGPU(L2 + 1, 1, 0);
		Atrans = MatrixGPU(_nAgent, L2 + 1, 0);
		Atrans.setTrans(&Ai);

		M.setBloc(0, _nAgent, 0, _nAgent, &H);
		M.setBloc(0, _nAgent, _nAgent, _Msize, &Atrans, -1);

			

		Rx1 = MatrixGPU(_nAgent, 1, 0); // Hx+q
		Rx2 = MatrixGPU(_nAgent, 1, 0); // -Ai^T*U

		Ru = MatrixGPU(L2 + 1, 1, 0); // Ru = W*(U-PI)

		U = MatrixGPU(L2 + 1, 1, 0);
		PI = MatrixGPU(L2 + 1, 1, 0);			

	}
	Pso = MatrixGPU(_nAgent, 1, 0, 1); // = Pn ? risque de non respect des contraintes	
	Pso.preallocateReduction();
	etaP = MatrixGPU(_nAgent, 1, 0, 1); 

	//std::cout << "autres donn�e sur GPU" << std::endl;
	
	
	//MatrixGPU temp1N(1, _nAgent, 0, 1);

	G2 = MatrixGPU(_nAgent, 0);
	initDCEndoMarket();
	// manque Ap3, Ap123, Bp3
	Ap3 = nVoisin;
	Ap123 = MatrixGPU(_nAgent, 1, 0, 1);
	Bp3 = MatrixGPU(_nAgent, 1, 0, 1); // 1/Mn * (Pso + P)/2 - eta/rho1
	
	Ap3.multiplyT(&nVoisin);
	Ap3.multiply(_rho1);
	Ap123.add(&Ap12, &Ap3);
	
	
	updateGlobalProbGPU();

	//Hosqp.display();
	
	//std::cout << " end init " << std::endl;
}



void ADMMGPUConstCons2::solveOPF()
{
	
	// update q
	MatrixGPU diffP(Pso);
	q.set(&etaP);
	diffP.add(&Pn);
	diffP.multiply(-_rho1/2);
	q.add(&diffP);
	
	//init
	int k = 0;
	float err = 2 * _epsIntern;
	mu = 10;
	MatrixGPU tempL21(L2 + 1, 1, 0);
	float valMin = 0.0000001;
	//boucle
	while (k< _iterIntern  && err> _epsIntern) {
	// update c
		
		c.linearOperation(&Ai, &Pso, &bi);
		
	// update PI
		for (int l = 0; l < L2; l++) {
			if (c.get(l, 0) < valMin) {
				PI.set(l, 0, mu / valMin); // eviter division par O
			}
			else {
				PI.set(l, 0, mu / c.get(l, 0)); // eviter division par O
			}
		}
		PI.set(L2, 0, -c.get(L2, 0) / mu);
	
	// update M
		// update Zvect
		
		Zvect.set(&U); 
		
		Zvect.set(L2, 0, 1); // egalite
		// update Z
		Z.setEyes(&Zvect);
		// update ZA
		ZA.multiplyMat(&Z, &Ai);
		// update W
		Wvect.set(&c);
		Wvect.set(L2, 0, mu);
		W.setEyes(&Wvect);

		M.setBloc(_nAgent, _Msize, 0, _nAgent, &ZA);
		M.setBloc(_nAgent, _Msize, _nAgent, _Msize, &W);
		try
		{
			Minv.invertGaussJordan(&M);
		}
		catch (const std::exception& e)
		{
			std::cout << e.what() << std::endl;
			
			std::cout << "k = " << k << " err= " << err << " alpha = " << alpha << " mu=" << mu << std::endl;
			c.display();
			Pn.display();
			Pso.display();
			std::cout << "---------------------------------" << std::endl;
			Pso.set(&Pn);
			return;
		}
		
	
	//update R
		// Rx
		Rx1.linearOperation(&H, &Pso, &q);
		Rx2.multiply(&Atrans, &U);
		Rx2.subtract(&Rx1);
		// Ru
		tempL21.subtract(&PI, &U);
		Ru.multiply(&W, &tempL21);

		R.setBloc(0, _nAgent, 0, 1, &Rx2);
		R.setBloc(_nAgent, _Msize, 0, 1, &Ru);
		//update pas
		pas.multiply(&Minv, &R);
		// find alpha
		findalpha();
		// update P, U
		pas.multiply(alpha);
		for (int n = 0; n < _nAgent; n++) {
			Pso.set(n, 0, Pso.get(n, 0) + pas.get(n, 0));
		}
		for (int l = 0; l < L2 + 1; l++) {
			U.set(l, 0, U.get(l, 0) + pas.get(_nAgent + l, 0));
		}

		// update mu
		mu *= 0.8;
		mu = MYMAX(mu, valMin);

		err = R.distance2();
		k++;
	}
	//std::cout << k << " " << err << std::endl;


}


void ADMMGPUConstCons2::findalpha()
{
	//version sur CPU
	alpha = 1;
	// U = U + alpha * pas >0
	for (int l = 0; l < L2; l++) {
		float div = pas.get(l + _nAgent, 0);
		if (div<0) {
			float newAlpha = -U.get(l, 0) / div;
			alpha = alpha < newAlpha ? alpha : newAlpha;
		}
	}
	// c > 0
	

	for (int l = 0; l < L2; l++) {
		float sum = 0;
		for (int p = 0; p < _nAgent; p++) {
			sum += Ai.get(l, p) * pas.get(p, 0);
		}
		if (sum < 0) {
			float newAlpha = -c.get(l, 0) / sum;
			alpha = alpha < newAlpha ? alpha : newAlpha;
		}
	}

	alpha *= 0.9;


}


void ADMMGPUConstCons2::solve(Simparam* result, const Simparam& sim, const StudyCase& cas)
{
#ifdef DEBUG_SOLVE
	cas.display();
	sim.display(1);
#endif // DEBUG_SOLVE
	
	
	tMarket = clock();
#ifdef INSTRUMENTATION
	std::chrono::high_resolution_clock::time_point t1;
	std::chrono::high_resolution_clock::time_point t2;
#endif // INSTRUMENTATION

	if (_id == 0) {
#ifdef INSTRUMENTATION
		hipDeviceSynchronize();
		t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION
		init(sim, cas);
#ifdef INSTRUMENTATION
		hipDeviceSynchronize();
		t2 = std::chrono::high_resolution_clock::now();
		timePerBlock.increment(0, 0, (float) std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
		occurencePerBlock.increment(0, 0, 1);
#endif // INSTRUMENTATION
	}
	//std::cout << _numBlocks2 << " " <<  _blockSize << std::endl;
	
	_at1 = _rhog; // represente en fait 2*a
	
	

	_resG = 2 * _epsG;
	float epsL2 = _epsL * _epsL;
	_iterGlobal = 0;
	
	
	//std::cout << iterG << " " << iterL << " " << epsL << " " << epsG << std::endl;
	while ((_iterGlobal < _iterG) && (_resG > _epsG)) {
#ifdef INSTRUMENTATION
		hipDeviceSynchronize();
		t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION

		updateLocalProbGPU(epsL2, _iterL);
#ifdef INSTRUMENTATION
		hipDeviceSynchronize();
		t2 = std::chrono::high_resolution_clock::now();
		timePerBlock.increment(0, 1, (float) std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
#endif // INSTRUMENTATION

		tradeLin.swap(&Tlocal); // echange juste les pointeurs	
		//std::cout << "-";
		
		updateGlobalProbGPU();
		
		if (!(_iterGlobal % _stepG)) {
#ifdef INSTRUMENTATION
			hipDeviceSynchronize();
			t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION
			_resG = updateRes(_iterGlobal / _stepG);
#ifdef INSTRUMENTATION
			hipDeviceSynchronize();
			t2 = std::chrono::high_resolution_clock::now();
			timePerBlock.increment(0, 8, (float) std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
#endif // INSTRUMENTATION
		}
		//std::cout << iterGlobal << " " << iterLocal << " " << resL << " " << resF.get(0, iterGlobal / stepG) << " " << resF.get(1, iterGlobal / stepG) << std::endl;
		_iterGlobal++;
	}
#ifdef INSTRUMENTATION
	occurencePerBlock.increment(0, 1, iterGlobal);
	occurencePerBlock.increment(0, 5, iterGlobal);
	occurencePerBlock.increment(0, 6, iterGlobal);
	occurencePerBlock.increment(0, 7, iterGlobal);
	occurencePerBlock.increment(0, 8, iterGlobal / stepG);

	hipDeviceSynchronize();
	t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION
	
	

	
	std::cout << "valeur finale des contraintes de l'opf : " << std::endl;
	c.display();
	//std::cout << iterGlobal << " " << iterLocal << " " << resL << " " << _resG << std::endl;
	
	setResult(result, cas.isAC());
	
#ifdef INSTRUMENTATION
	hipDeviceSynchronize();
	t2 = std::chrono::high_resolution_clock::now();
	timePerBlock.increment(0, 9, (float) std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
	occurencePerBlock.increment(0, 9, 1);
	result->setTimeBloc(&timePerBlock, &occurencePerBlock);
#endif // INSTRUMENTATION

}

void ADMMGPUConstCons2::updateLocalProbGPU(float epsL, int nIterL) {
	int numBlocks = _nAgent;
	switch (_blockSize) {
	case 512:
		updateTradePGPUSharedResidualCons<512> << <numBlocks, _blockSize >> > (Tlocal._matrixGPU, Tlocal_pre._matrixGPU, Tmoy._matrixGPU, P._matrixGPU, MU._matrixGPU, nVoisin._matrixGPU, _at1, _at2, Bt1._matrixGPU, Ct._matrixGPU,
			matLb._matrixGPU, matUb._matrixGPU, Ap1._matrixGPU, Ap3._matrixGPU, Ap123._matrixGPU, Bp3._matrixGPU, Cp._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, CoresAgentLin._matrixGPU, epsL, nIterL);
		break;
	case 256:
		updateTradePGPUSharedResidualCons<256> << <numBlocks, _blockSize >> > (Tlocal._matrixGPU, Tlocal_pre._matrixGPU, Tmoy._matrixGPU, P._matrixGPU, MU._matrixGPU, nVoisin._matrixGPU, _at1, _at2, Bt1._matrixGPU, Ct._matrixGPU,
			matLb._matrixGPU, matUb._matrixGPU, Ap1._matrixGPU, Ap3._matrixGPU, Ap123._matrixGPU, Bp3._matrixGPU, Cp._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, CoresAgentLin._matrixGPU, epsL, nIterL);
		break;
	case 128:
		updateTradePGPUSharedResidualCons<128> << <numBlocks, _blockSize >> > (Tlocal._matrixGPU, Tlocal_pre._matrixGPU, Tmoy._matrixGPU, P._matrixGPU, MU._matrixGPU, nVoisin._matrixGPU, _at1, _at2, Bt1._matrixGPU, Ct._matrixGPU,
			matLb._matrixGPU, matUb._matrixGPU, Ap1._matrixGPU, Ap3._matrixGPU, Ap123._matrixGPU, Bp3._matrixGPU, Cp._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, CoresAgentLin._matrixGPU, epsL, nIterL);
		break;
	case 64:
		updateTradePGPUSharedResidualCons< 64> << <numBlocks, _blockSize >> > (Tlocal._matrixGPU, Tlocal_pre._matrixGPU, Tmoy._matrixGPU, P._matrixGPU, MU._matrixGPU, nVoisin._matrixGPU, _at1, _at2, Bt1._matrixGPU, Ct._matrixGPU,
			matLb._matrixGPU, matUb._matrixGPU, Ap1._matrixGPU, Ap3._matrixGPU, Ap123._matrixGPU, Bp3._matrixGPU, Cp._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, CoresAgentLin._matrixGPU, epsL, nIterL);
		break;
	case 32:
		updateTradePGPUSharedResidualCons< 32> << <numBlocks, _blockSize >> > (Tlocal._matrixGPU, Tlocal_pre._matrixGPU, Tmoy._matrixGPU, P._matrixGPU, MU._matrixGPU, nVoisin._matrixGPU, _at1, _at2, Bt1._matrixGPU, Ct._matrixGPU,
			matLb._matrixGPU, matUb._matrixGPU, Ap1._matrixGPU, Ap3._matrixGPU, Ap123._matrixGPU, Bp3._matrixGPU, Cp._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, CoresAgentLin._matrixGPU, epsL, nIterL);
		break;
	case 16:
		updateTradePGPUSharedResidualCons< 16> << <numBlocks, _blockSize >> > (Tlocal._matrixGPU, Tlocal_pre._matrixGPU, Tmoy._matrixGPU, P._matrixGPU, MU._matrixGPU, nVoisin._matrixGPU, _at1, _at2, Bt1._matrixGPU, Ct._matrixGPU,
			matLb._matrixGPU, matUb._matrixGPU, Ap1._matrixGPU, Ap3._matrixGPU, Ap123._matrixGPU, Bp3._matrixGPU, Cp._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, CoresAgentLin._matrixGPU, epsL, nIterL);
		break;
	case  8:
		updateTradePGPUSharedResidualCons<  8> << <numBlocks, _blockSize >> > (Tlocal._matrixGPU, Tlocal_pre._matrixGPU, Tmoy._matrixGPU, P._matrixGPU, MU._matrixGPU, nVoisin._matrixGPU, _at1, _at2, Bt1._matrixGPU, Ct._matrixGPU,
			matLb._matrixGPU, matUb._matrixGPU, Ap1._matrixGPU, Ap3._matrixGPU, Ap123._matrixGPU, Bp3._matrixGPU, Cp._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, CoresAgentLin._matrixGPU, epsL, nIterL);
		break;
	case  4:
		updateTradePGPUSharedResidualCons<  4> << <numBlocks, _blockSize >> > (Tlocal._matrixGPU, Tlocal_pre._matrixGPU, Tmoy._matrixGPU, P._matrixGPU, MU._matrixGPU, nVoisin._matrixGPU, _at1, _at2, Bt1._matrixGPU, Ct._matrixGPU,
			matLb._matrixGPU, matUb._matrixGPU, Ap1._matrixGPU, Ap3._matrixGPU, Ap123._matrixGPU, Bp3._matrixGPU, Cp._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, CoresAgentLin._matrixGPU, epsL, nIterL);
		break;
	case  2:
		updateTradePGPUSharedResidualCons<  2> << <numBlocks, _blockSize >> > (Tlocal._matrixGPU, Tlocal_pre._matrixGPU, Tmoy._matrixGPU, P._matrixGPU, MU._matrixGPU, nVoisin._matrixGPU, _at1, _at2, Bt1._matrixGPU, Ct._matrixGPU,
			matLb._matrixGPU, matUb._matrixGPU, Ap1._matrixGPU, Ap3._matrixGPU, Ap123._matrixGPU, Bp3._matrixGPU, Cp._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, CoresAgentLin._matrixGPU, epsL, nIterL);
		break;
	case  1:
		updateTradePGPUSharedResidualCons<  1> << <numBlocks, _blockSize >> > (Tlocal._matrixGPU, Tlocal_pre._matrixGPU, Tmoy._matrixGPU, P._matrixGPU, MU._matrixGPU, nVoisin._matrixGPU, _at1, _at2, Bt1._matrixGPU, Ct._matrixGPU,
			matLb._matrixGPU, matUb._matrixGPU, Ap1._matrixGPU, Ap3._matrixGPU, Ap123._matrixGPU, Bp3._matrixGPU, Cp._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, CoresAgentLin._matrixGPU, epsL, nIterL);
		break;
	}
	//hipStreamSynchronize(streamCalculation);
}



void ADMMGPUConstCons2::updateGlobalProbGPU()
{
	//Rem : tout calcul qui est de taille N ou M peut �tre fait par les agents
		// Si le calcul est de taile L, soit c'est calcul� par un/des superviseurs, soit tous les agents le calcul (un peu absurde)

#ifdef INSTRUMENTATION
// FB 3a
	hipDeviceSynchronize();
	std::chrono::high_resolution_clock::time_point t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION


	updatePnGPU << <_numBlocksN, _blockSize >> > (Pn._matrixGPU, Tmoy._matrixGPU, nVoisin._matrixGPU, _nAgent);
	
#ifdef INSTRUMENTATION
	hipDeviceSynchronize();
	std::chrono::high_resolution_clock::time_point t2 = std::chrono::high_resolution_clock::now();

	timePerBlock.increment(0, 5, (float) std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());

	// FB 3b
	hipDeviceSynchronize();
	t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION	

	// Resolution de l'OPF
	if (_nLine) {
		
		Pso.transferCPU();
		Pn.transferCPU();
		etaP.transferCPU();
		/*std::cout << " Pn :" << std::endl;
		Pn.display();
		std::cout << " etaP :" << std::endl;
		etaP.display();*/

		solveOPF();
		/*std::cout << " Pso :" << std::endl;
		Pso.display();*/
		Pn.transferGPU();
		etaP.transferGPU();
		Pso.transferGPU();
		
	}
	else {
		Pso = Pn;
	}
	
	
	

#ifdef INSTRUMENTATION
	hipDeviceSynchronize();
	t2 = std::chrono::high_resolution_clock::now();
	timePerBlock.increment(0, 6, (float) std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());

	// FB 3c
	hipDeviceSynchronize();
	t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION

	// update Bp3
	updateEtaPBp3 << <_numBlocksN, _blockSize >> > (Bp3._matrixGPU, etaP._matrixGPU, nVoisin._matrixGPU, Pso._matrixGPU, Pn._matrixGPU, _rho1, _nAgent);
	updateLAMBDABt1GPU << <_numBlocksM, _blockSize >> > (Bt1._matrixGPU, LAMBDALin._matrixGPU, tradeLin._matrixGPU, _rhog, CoresLinTrans._matrixGPU, _nTrade);
	

#ifdef INSTRUMENTATION
	hipDeviceSynchronize();
	t2 = std::chrono::high_resolution_clock::now();
	timePerBlock.increment(0, 7, (float) std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
#endif // INSTRUMENTATION
}



float ADMMGPUConstCons2::updateResEndo(int iter)
{

	float resS = Tlocal.max2(&tradeLin);

	updateDiffGPU <<<_numBlocksM, _blockSize >>> (tempNN._matrixGPU, Tlocal._matrixGPU, CoresLinTrans._matrixGPU, _nAgent);
	float resR = tempNN.max2();

	float resXf = _ratioEps * Pso.max2(&Pn);
	resF.set(0, iter, resR);
	resF.set(1, iter, resS);
	resF.set(2, iter, resXf);
	return MYMAX(MYMAX(resXf, resS), resR);

}





