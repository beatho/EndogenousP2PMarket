#include "hip/hip_runtime.h"
#pragma once
#include "../head/MethodP2P.cuh"
#define MAX(X, Y) X * (X >= Y) + Y * (Y > X)


MethodP2P::MethodP2P() : Method()
{
#if DEBUG_CONSTRUCTOR
	std::cout << "method constructor" << std::endl;
#endif // DEBUG_CONSTRUCTOR
	timePerBlock = MatrixCPU(1, 9, 0); // Fb0, Fb1abcd, Fb2, Fb3, Fb5, Fb0'
	// si les sous ensemble ne sont pas accessible, tout est dans le premier.
	occurencePerBlock = MatrixCPU(1, 9, 0); //nb de fois utilis� pendant la simu
}

MethodP2P::~MethodP2P() 
{
}

void MethodP2P::updateLAMBDA(MatrixCPU* LAMBDA, MatrixCPU* trade, float rho)
{
	MatrixCPU times = MatrixCPU(*trade);
	times.addTrans(trade);
	times.multiply(rho);
	times.multiply(0.5);
	LAMBDA->add(LAMBDA, &times);
}

float MethodP2P::updateRes(MatrixCPU* res, MatrixCPU* Tlocal, MatrixCPU* trade, int iter)
{
	MatrixCPU temp(*Tlocal);
	temp.addTrans(Tlocal);

	MatrixCPU temp2(*Tlocal);
	float resR = temp.max2();
	temp2.subtract(trade);
	float resS = temp2.max2();

	res->set(0, iter, resR);
	res->set(1, iter, resS);

	return resR * (resR > resS) + resS * (resR <= resS);

}

float MethodP2P::updateRes(MatrixCPU* res, MatrixCPU* Tlocal, MatrixCPU* trade, int iter, MatrixCPU* Kappa1, MatrixCPU* Kappa2, MatrixCPU* Kappa1_pre, MatrixCPU* Kappa2_pre)
{
	MatrixCPU temp(*Tlocal);
	temp.addTrans(Tlocal);

	MatrixCPU temp2(*Tlocal);
	float resR = temp.max2();
	temp2.subtract(trade);
	float resS = temp2.max2();

	MatrixCPU tempL(*Kappa1);
	MatrixCPU tempL2(*Kappa2);
	Kappa1_pre->projectNeg();
	Kappa2_pre->projectNeg();
	tempL.projectNeg();
	tempL2.projectNeg();
	tempL.subtract(Kappa1_pre);
	tempL2.subtract(Kappa2_pre);
	tempL.multiplyT(&tempL);
	tempL2.multiplyT(&tempL2);
	tempL.add(&tempL2);

	float resX = _ratioEps * sqrt(tempL.max2());
	
	res->set(0, iter, resR);
	res->set(1, iter, resS);
	res->set(2, iter, resX);

	return MAX(MAX(resX,resS),resR);
}




void MethodP2P::updateLAMBDA(MatrixGPU* LAMBDA, MatrixGPU* trade, float rho, MatrixGPU* tempNN)
{
	tempNN->set(trade);
	tempNN->addTrans(trade);
	tempNN->multiply(rho);
	tempNN->multiply(0.5);
	LAMBDA->add(LAMBDA, tempNN);
}

void MethodP2P::updateKappa(MatrixCPU* Kappa1, MatrixCPU* Kappa2, MatrixCPU* L, MatrixCPU* Qtot)
{
	
	
	//
	Kappa1->projectNeg();
	Kappa1->add(L);
	Kappa1->subtract(Qtot);
	
	Kappa2->projectNeg();
	Kappa2->add(L);
	Kappa2->add(Qtot);
	//
	
}

void MethodP2P::updateKappa(MatrixGPU* Kappa1, MatrixGPU* Kappa2, MatrixGPU* L, MatrixGPU* Qtot)
{
	Kappa1->projectNeg();
	Kappa1->add(L);
	Kappa1->subtract(Qtot);
	Kappa2->projectNeg();
	Kappa2->add(L);
	Kappa2->add(Qtot);
}

void MethodP2P::updateCp2(MatrixCPU* Cp2, float rho1, MatrixCPU* Kappa1, MatrixCPU* Kappa2, MatrixCPU* G, MatrixCPU* tempL1, MatrixCPU* Qpart, MatrixCPU* nVoisin, int nLine, int nAgent)
{
	tempL1->subtractAbs(Kappa1, Kappa2);
	//Cp2->multiplyTrans(G, tempL1, 0);

	float r = 0;
	for (int i = 0; i < nAgent; ++i)
	{
		r = 0;
		for (int k = 0; k < nLine; ++k)
		{
			r +=  G->get(k, i) * (tempL1->get(k, 0) + 2 * Qpart->get(k, i));
		}
		Cp2->set(i, 0, r);
	}

	Cp2->multiply(rho1);
	Cp2->multiplyT(nVoisin);
}

float MethodP2P::updateRes(MatrixCPU* res, MatrixGPU* Tlocal, MatrixGPU* trade, int iter, MatrixGPU* tempNN)
{
	tempNN->subtract(Tlocal, trade);
	
	float resS = tempNN->max2();
	tempNN->set(Tlocal);
	tempNN->addTrans(Tlocal);
	float resR = tempNN->max2();
	

	res->set(0, iter, resR);
	res->set(1, iter, resS);
	

	return resR* (resR > resS) + resS * (resR <= resS);
}





void MethodP2P::updatePn(MatrixCPU* Pn, MatrixCPU* Tmoy, MatrixCPU* nVoisin)
{
	Pn->set(Tmoy);
	Pn->multiplyT(nVoisin);
}
void MethodP2P::updatePn(MatrixGPU* Pn, MatrixGPU* Tmoy, MatrixGPU* nVoisin)
{
	Pn->set(Tmoy);
	Pn->multiplyT(nVoisin);
}
void MethodP2P::updatePn(MatrixCPU* Pn, MatrixCPU* trade)
{
	Pn->sum(trade);
}

void MethodP2P::solveWithMinPower(Simparam* result, const Simparam& sim, const StudyCase& cas)
{
	std::cout << "solveWithMinPower : should not be called" << std::endl;
}




void MethodP2P::resetId()
{
	_id = 0;
}



__global__ void updateLAMBDAGPU(float* LAMBDALin, float* tradeLin, float rho, float* CoresLinTrans, int const N)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int step = blockDim.x * gridDim.x;
	for (int l = index; l < N; l += step)
	{
		float m = LAMBDALin[l];
		int k = CoresLinTrans[l];
		LAMBDALin[l] = m + 0.5 * rho * (tradeLin[l] + tradeLin[k]);
	}
}
__global__ void updateBt1GPU(float* Bt1, float* tradeLin, float rho, float* LAMBDA, float* CoresLinTrans, int const N)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int step = blockDim.x * gridDim.x;
	for (int l = index; l < N; l += step)
	{
		int k = CoresLinTrans[l];
		Bt1[l] = 0.5 * (tradeLin[l] - tradeLin[k]) - LAMBDA[l] / rho;
	}

}

__global__ void updateLAMBDABt1GPU(float* Bt1, float* LAMBDA, float* tradeLin, float rho, float* CoresLinTrans, int const N) {

	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int step = blockDim.x * gridDim.x;
	for (int l = index; l < N; l += step)
	{
		int k = CoresLinTrans[l];
		float m = LAMBDA[l] + 0.5 * rho * (tradeLin[l] + tradeLin[k]);
		Bt1[l] = 0.5 * (tradeLin[l] - tradeLin[k]) - m / rho;
		LAMBDA[l] = m;
	}
}


__global__ void updateDiffGPU(float* tempN, float* Tlocal, float* CoresLinTrans, int const N)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int step = blockDim.x * gridDim.x;
	for (int l = index; l < N; l += step)
	{
		int k = CoresLinTrans[l];
		tempN[l] = (Tlocal[l] + Tlocal[k]);
	}
}

__global__ void updateResKappa(float* result, float* Kappa1, float* Kappa2, float* Kappapre1, float* Kappapre2 ,float ratio, int const L)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int step = blockDim.x * gridDim.x;
	for (int l = index; l < L; l += step)
	{
		float kappaNeg1 = Kappa1[l] < 0 ? Kappa1[l] : 0;
		float kappaNeg2 = Kappa2[l] < 0 ? Kappa2[l] : 0;
		float kappaNegpre1 = Kappapre1[l] < 0 ? Kappapre1[l] : 0;
		float kappaNegpre2 = Kappapre2[l] < 0 ? Kappapre2[l] : 0;

		float res1 = kappaNeg1 - kappaNegpre1;
		res1 *= res1;
		float res2 = kappaNeg2 - kappaNegpre2;
		res2 *= res2;

		result[l] = ratio * sqrt(res1 + res2);
	}
}


__global__ void selectResidual(float* res, unsigned int id1, unsigned int id2, unsigned int id3, float* output) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index == 0) {
		float max = res[id1] > res[id2] ? res[id1] : res[id2];
		max = res[id3] > max ? res[id3] : max;
		*output = max;
	}
}

__global__ void updateKappaGPU(float* Kappa1, float* Kappa2, float* Llimit, float* Qtot, int nLine)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int step = blockDim.x * gridDim.x;
	for (int l = index; l < nLine; l += step)
	{
		float kappaNeg1 = Kappa1[l] < 0 ? Kappa1[l] : 0;
		float kappaNeg2 = Kappa2[l] < 0 ? Kappa2[l] : 0;
		float lim = Llimit[l];
		float Q = Qtot[l];
		Kappa1[l] = kappaNeg1 + lim - Q;
		Kappa2[l] = kappaNeg2 + lim + Q;
	}
}
__global__ void diffKappa(float* tempL1, float* Kappa1, float* Kappa2, int nLine)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int step = blockDim.x * gridDim.x;
	for (int l = index; l < nLine; l += step)
	{
		float Kappa1Abs = Kappa1[l] > 0 ? Kappa1[l] : -Kappa1[l]; //2 * (Kappa1[l] > 0) * Kappa1[l] - Kappa1[l]; // Kappa1[l] > 0 ? Kappa1[l] : -Kappa1[l]
		float Kappa2Abs = Kappa2[l] > 0 ? Kappa2[l] : -Kappa2[l]; //2 * (Kappa2[l] > 0) * Kappa2[l] - Kappa2[l]; // Kappa2[l] > 0 ? Kappa2[l] : -Kappa2[l]
		tempL1[l] = Kappa1Abs - Kappa2Abs;
	}
}


__global__ void updateCpOld(float* Cp, float* Cp1, float* Cp2, float* tempN1, float* nVoisin, const float rho1, const int nAgent) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int step = blockDim.x * gridDim.x;
	for (int n = index; n < nAgent; n += step)
	{
		Cp[n] = Cp1[n] + rho1 * (nVoisin[n] * (Cp2[n] + tempN1[n]));
	}
}

__global__ void updateCp(float* Cp, float* Cp1, float* Cp2, const int nAgent) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int step = blockDim.x * gridDim.x;
	for (int n = index; n < nAgent; n += step)
	{
		Cp[n] = Cp1[n] + Cp2[n];
	}
}

__global__ void updateQpart(float* Qpart, float* alpha, const int nAgent) {
	int index = threadIdx.x;
	int step = blockDim.x;
	int l = blockIdx.x;
	extern __shared__ float shAlpha[];

	for (int n = index; n < nAgent; n += step)
	{
		shAlpha[n] = alpha[l * nAgent + n];
	}
	__syncthreads();

	float s_pre = 0;
	int n_pre = nAgent - 1;
	for (int n = (nAgent - index - 1); n >= 0; n -= step)
	{
		float s = 0;
		for (int j = n_pre; j > n; j--) {
			s += shAlpha[j]; // c'est moche cet acc�s de m�moire partag�e
		}
		s = s + s_pre;
		Qpart[l * nAgent + n] = s;
		s_pre = s;
		n_pre = n;
	}
}
__global__ void updateQpartTrans(float* Qpart, float* alpha, const int N, const int nLine) {
	int index = threadIdx.x;
	int step = blockDim.x;
	int l = blockIdx.x;
	extern __shared__ float shAlpha[];

	for (int n = index; n < N; n += step)
	{
		shAlpha[n] = alpha[n * nLine + l]; // moche
	}
	__syncthreads();
	float s_pre = 0;
	int n_pre = N - 1;
	for (int n = (N - index - 1); n >= 0; n -= step)
	{
		float s = 0;
		for (int j = n_pre; j > n; j--) {
			s += shAlpha[j]; // c'est moche cet acc�s de m�moire partag�e
		}
		s = s + s_pre;
		Qpart[n * nLine + l] = s;
		s_pre = s;
		n_pre = n;
	}
}


__global__ void updateQtot(float* Qtot, float* Qpart, float* alpha, const int nLine, const int nAgent) {


	int thIdx = threadIdx.x + blockIdx.x * blockDim.x;
	int step = blockDim.x * gridDim.x;

	for (int l = thIdx; l < nLine; l += step) {

		Qtot[l] = Qpart[l * nAgent] + alpha[l * nAgent];
	}
}
__global__ void updateQtotTrans(float* Qtot, float* Qpart, float* alpha, const int nLine) {


	int thIdx = threadIdx.x + blockIdx.x * blockDim.x;
	int step = blockDim.x * gridDim.x;

	for (int l = thIdx; l < nLine; l += step) {

		Qtot[l] = Qpart[l] + alpha[l];
	}
}

__global__ void updateAlpha(float* alpha, float* G, float* Pn, const int nLine, const int nAgent)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	int step = blockDim.x * gridDim.x;
	int N = nAgent * nLine;

	for (int i = index; i < N; i += step)
	{
		int k = i % nAgent;
		alpha[i] = G[i] * Pn[k];
	}
}
__global__ void updateAlphaTrans(float* alpha, float* GTrans, float* Pn, const int nLine, const int nAgent) {

	int index = threadIdx.x + blockIdx.x * blockDim.x;
	int step = blockDim.x * gridDim.x;
	int N = nAgent * nLine;

	for (int i = index; i < N; i += step)
	{
		int k = i / nLine;
		alpha[i] = GTrans[i] * Pn[k];
	}

}


__global__ void updateResX(float* res, float* Kappa1, float* Kappa2, float* KappaPre1, float* KappaPre2, const int nLine) {

	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int step = blockDim.x * gridDim.x;
	for (int l = index; l < nLine; l += step)
	{
		float k1 = (Kappa1[l] < 0) * Kappa1[l];
		float k2 = (Kappa2[l] < 0) * Kappa2[l];
		float k1pre = (KappaPre1[l] < 0) * KappaPre1[l];
		float k2pre = (KappaPre2[l] < 0) * KappaPre2[l];

		k1 -= k1pre;
		k2 -= k2pre;

		res[l] = k1 * k1 + k2 * k2;
	}
}



__global__ void updatePnGPU(float* Pn, float* Tmoy, float* nVoisin, const int nAgent)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int step = blockDim.x * gridDim.x;
	for (int n = index; n < nAgent; n += step)
	{
		Pn[n] = Tmoy[n] * nVoisin[n];
	}

}



/*
__global__ void updateUAiq(float* UAiq, float* u, float* Aiq, int N, int size) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int step = blockDim.x * gridDim.x;
	for (int n = index; n < size; n += step)
	{
		int row = n / N;
		UAiq[n] = u[row] * Aiq[n];
	}
}

// Ru = U*g + epsi
__global__ void updateRu(float* Ru, float* U, float* g, float epsi, int N, int L2) {

	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int step = blockDim.x * gridDim.x;
	for (int i = index; i < L2; i += step)
	{
		
		Ru[i + N] = U[i] * g[i] + epsi;
	}

}

__global__ void updateV(float* v, float* pas, float* alpha, int offset) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index == 0) {
		*v = *v + (*alpha) * pas[offset];
	}
}

__global__ void updateQt(float* qt, float* Pso, float* Pn, float* etaSO, float rho1, int N) {

	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int step = blockDim.x * gridDim.x;
	for (int i = index; i < N; i += step)
	{

		qt[i] = etaSO[i] - rho1 * (Pso[i] + Pn[i]) / 2;
	}


}*/

__global__ void updatePI(float* PI, float* c, float mu, float valMin, int L) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int step = blockDim.x * gridDim.x;
	for (int i = index; i < L; i += step)
	{
		PI[i] = c[i] < valMin ? mu / valMin : mu / c[i];
	}
	if (index == 0) {
		PI[L] = -c[L] / mu;
	}
}


__global__ void updatePso(float* Pso, float* pas, float* alpha, int N) {

	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int step = blockDim.x * gridDim.x;
	for (int i = index; i < N; i += step)
	{

		Pso[i] = Pso[i] + *alpha * pas[i];
	}

}

__global__ void updateU(float* U, float* pas, float* alpha, int N, int L2) {

	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int step = blockDim.x * gridDim.x;
	for (int i = index; i < L2; i += step)
	{

		U[i] = U[i] + (*alpha) * pas[i + N];
	}

}


__global__ void updateEtaPBp3(float* Bp3, float* etaP, float* nVoisin, float* Pso, float* Pn, float rho, const int nAgent) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int step = blockDim.x * gridDim.x;
	for (int l = index; l < nAgent; l += step)
	{
		
		float m = etaP[l] + 0.5 * rho * (Pso[l] - Pn[l]);
		Bp3[l] = (0.5 * (Pso[l] + Pn[l]) + m / rho) / nVoisin[l];
		etaP[l] = m;
	}


}

