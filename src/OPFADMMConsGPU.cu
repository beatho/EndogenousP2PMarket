#include "hip/hip_runtime.h"
﻿#include "../head/OPFADMMConsGPU.cuh"
 


OPFADMMConsGPU::OPFADMMConsGPU() : MethodOPFGPU()
{
#if DEBUG_CONSTRUCTOR
	std::cout << " OPFADMMConsGPU Constructor" << std::endl;
#endif // DEBUG_CONSTRUCTOR
	_name = NAME;
	timePerBlock = MatrixCPU(1, 12, 0); // Fb0, Fb11abcd, FB12, Fb2, Fb3, Fb4, Fb5,FB6, Fb0'
	// si les sous ensemble ne sont pas accessible, tout est dans le premier.
	occurencePerBlock = MatrixCPU(1, 12, 0); //nb de fois utilisé pendant la simu
}


OPFADMMConsGPU::OPFADMMConsGPU(float rho) : MethodOPFGPU()
{
#if DEBUG_CONSTRUCTOR
	std::cout << "default OPFADMMConsGPU Constructor" << std::endl;
#endif // DEBUG_CONSTRUCTOR
	_name = NAME;
	_rho = rho;
	timePerBlock = MatrixCPU(1, 12, 0); // Fb0, Fb11, FB12, Fb2, Fb3, Fb4, Fb5, FB6, Fb0'
	// si les sous ensemble ne sont pas accessible, tout est dans le premier.
	occurencePerBlock = MatrixCPU(1, 12, 0); //nb de fois utilisé pendant la simu
}

OPFADMMConsGPU::~OPFADMMConsGPU()
{
	 
}
void OPFADMMConsGPU::setParam(float rho)
{
	_rho = rho;
}

bool OPFADMMConsGPU::chekcase()
{
	if (_nBus != (_nLine + 1)) {
		std::cout << "wrong number of line " << _nLine << "against " << _nBus << std::endl;
		return false;
	}
	//CoresLineBus.display();
	for (int i = 0; i < _nLine; i++) {
		if (CoresLineBusCPU.get(i, 1) != (i + 1)) {
			std::cout << "wrong numerotation of line " << CoresLineBusCPU.get(i, 1) << "against " << (i + 1) << std::endl;
			return false;
		}
		if (CoresLineBusCPU.get(i, 0) > CoresLineBusCPU.get(i, 1)) {
			std::cout << "wrong numeoration of bus " << CoresLineBusCPU.get(i, 0) << "against " << CoresLineBusCPU.get(i, 1) << std::endl;
			return false;
		}
	}
	if (ZsRe.getNLin() == 0  || ZsIm.getNLin() == 0) {
		std::cout << "matrice non defined, ZsRe, Zs Im, Yd" << std::endl;
		ZsRe.display(true);
		ZsIm.display(true);
		return false;
	}

	return true;
}

void OPFADMMConsGPU::solve(Simparam* result, const Simparam& sim, const StudyCase& cas)
{
#ifdef DEBUG_SOLVE
	cas.display();
	sim.display(1);
#endif // DEBUG_SOLVE
	
	clock_t tall =clock();
#ifdef INSTRUMENTATION
	std::chrono::high_resolution_clock::time_point t1;
	std::chrono::high_resolution_clock::time_point t2;
#endif // INSTRUMENTATION

	// FB 0
	if (_id == 0) {
#ifdef INSTRUMENTATION
		t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION
		init(sim, cas);
#ifdef INSTRUMENTATION
		t2 = std::chrono::high_resolution_clock::now();
		timePerBlock.increment(0, 0, (float) std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
		occurencePerBlock.increment(0, 0, 1);
#endif // INSTRUMENTATION
	}
	
	_iterG = sim.getIterG();
	int iterL = sim.getIterL();
	_stepG = sim.getStepG();
	int stepL = sim.getStepL();
	
	float epsG = sim.getEpsG();
	float epsL = sim.getEpsL();
	
	
	float fc = 0;
	float resG = 2 * epsG;
	_iterGlobal = 0;
	

	//Chat.display();
	//Bpt2.display();
	
	while ((_iterGlobal < _iterG) && (resG>epsG)) {
		
		
		updateXWOCurrent();


		// 
		//updateXWOCurrentOnCPU();
		//updateXWOCurrentOnCPUBis();
		CHECK_LAST_CUDA_ERROR();
#ifdef INSTRUMENTATION
		t2 = std::chrono::high_resolution_clock::now();
		timePerBlock.increment(0, 5, (float) std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
		t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION
		
		CommunicationX();
		
#ifdef INSTRUMENTATION
		t2 = std::chrono::high_resolution_clock::now();
		timePerBlock.increment(0, 6, (float) std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
		t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION
		updateGlobalProb();
		CHECK_LAST_CUDA_ERROR();
		updateMu();
		CHECK_LAST_CUDA_ERROR();
#ifdef INSTRUMENTATION
		t2 = std::chrono::high_resolution_clock::now();
		timePerBlock.increment(0, 7, (float) std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
		t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION
		updateChat();

		CHECK_LAST_CUDA_ERROR();
#ifdef INSTRUMENTATION
		t2 = std::chrono::high_resolution_clock::now();
		timePerBlock.increment(0, 8, (float) std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
#endif // INSTRUMENTATION
		// FB 4
		if (!(_iterGlobal % _stepG)) {
#ifdef INSTRUMENTATION
			t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION
			resG = updateRes(_iterGlobal / _stepG);
			//std::cout << _iterGlobal << " " << _iterLocal << " " << resL << " " << resF.get(0, _iterGlobal / _stepG) << " " << resF.get(1, _iterGlobal / _stepG) << std::endl;
			//resG = 1;
#ifdef INSTRUMENTATION
			t2 = std::chrono::high_resolution_clock::now();
			timePerBlock.increment(0, 9, (float) std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
#endif // INSTRUMENTATION
		}
		//std::cout << iterGlobal << " " << _iterLocal << " " << resL << " " << resF.get(0, iterGlobal / stepG) << " " << resF.get(1, iterGlobal / stepG) << std::endl;

		_iterGlobal++;
	}
	std::cout << "---------------------------------" << std::endl;
		/*std::cout << " X " << std::endl;
		X.display(true);
		std::cout << " Chat " << std::endl;
		Chat.display(true);
		std::cout << " Q " << std::endl;
		Q.display(true);
		std::cout << "Y " << std::endl;
		Y.display(true);
		std::cout << "Mu " << std::endl;
		Mu.display(true);
		*/
	std::cout << _iterGlobal << " " << resF.get(0, (_iterGlobal - 1) / _stepG) << " " << resF.get(1, (_iterGlobal - 1) / _stepG) << " " << resF.get(2, (_iterGlobal - 1) / _stepG) << std::endl;


#ifdef INSTRUMENTATION	
	occurencePerBlock.increment(0, 5, _iterGlobal);
	occurencePerBlock.increment(0, 6, _iterGlobal);
	occurencePerBlock.increment(0, 7, _iterGlobal);
	occurencePerBlock.increment(0, 8, _iterGlobal);
	occurencePerBlock.increment(0, 9, _iterGlobal / _stepG);

	t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION

	//setPnFromX << < _nBus, _blockSizeSmall >> > (Pn._matrixGPU, X._matrixGPU, _indiceBusBegin._matrixGPU, _CoresAgentBus._matrixGPU, _nAgentByBus._matrixGPU, _CoresAgentBusBegin._matrixGPU, _nAgent);
	////CHECK_LAST_CUDA_ERROR();
	MatrixCPU PnCPU;
	Pn.toMatCPU(PnCPU);
	PnCPU.set(0, 0, getPLoss());
	PnCPU.set(_nAgent, 0, getQLoss());
	
	fc = calcFc(&Cost1, &Cost2, &Pn, &tempN2);
	// FB 5
	
	result->setResF(&resF);
	
	/*std::cout << "--------" << std::endl;
	std::cout << " Pn " << std::endl;
	Pn.display();
	for (int i = 0; i < 3; i++) {
		std::cout << " X " << i << std::endl;
		X[i].display();
	}
	*/
	MatrixCPU Pb(getPb());
	MatrixCPU Phi(getPhi());
	MatrixCPU E(getE());

	result->setE(&E);
	result->setPhi(&Phi);
	result->setPb(&Pb);
	

	result->setIter(_iterGlobal);
	

	result->setPn(&PnCPU);
	
	result->setFc(fc);

#ifdef INSTRUMENTATION
	t2 = std::chrono::high_resolution_clock::now();
	timePerBlock.increment(0, 10, (float) std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
	occurencePerBlock.increment(0, 10, 1);

	result->setTimeBloc(&timePerBlock, &occurencePerBlock);
#endif // INSTRUMENTATION
	tall = clock() - tall;
	timeOPF = tall;

	result->setTime((float)tall / CLOCKS_PER_SEC);
	 

}

void OPFADMMConsGPU::updateP0(const StudyCase& cas)
{
	_id = _id + 1;
#ifdef INSTRUMENTATION
	std::chrono::high_resolution_clock::time_point t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION
    
	Pmin = cas.getPmin();
	Pmax = cas.getPmax();
	Cost2 = cas.getb();

	// pour essayer que cela marche
	Pn.add(&Pmin, &Pmax);
	Pn.divide(2);
	
	// remove loss agent
	Pn.set(0, 0, 0, 1);
	Pmin.set(0, 0, 0, 1);
	Pmax.set(0, 0, 0, 1);
	Pn.set(_nAgent, 0, 0, 1);
	Pmin.set(_nAgent, 0, 0, 1);
	Pmax.set(_nAgent, 0, 0, 1);
	
	
	ComputePFromAgentToBus();

	initPQAgent << < _nBus, _blockSize >> > (X._matrixGPU, _indiceBusBegin._matrixGPU, _CoresAgentBus._matrixGPU, _nAgentByBus._matrixGPU, _CoresAgentBusBegin._matrixGPU, Pn._matrixGPU, _nAgent);

	//_global__ void initDFSPQ(float* X, float* Pb, float* nChild, float* Childs, float* indiceBusBegin, float* indiceChildBegin, int nBus)
	initDFSPQ << <1, _nBus, _nBus* (sizeof(bool) + sizeof(int)) >> > (X._matrixGPU, Pb._matrixGPU, nChild._matrixGPU, Childs._matrixGPU, _indiceBusBegin._matrixGPU, _indiceChildBegin._matrixGPU, _nBus);
	communicateX << <_nBus, _blockSize >> > (X._matrixGPU, nChild._matrixGPU, Ancestor._matrixGPU, Childs._matrixGPU, _indiceBusBegin._matrixGPU, _indiceChildBegin._matrixGPU, _nAgentByBus._matrixGPU, _nBus);


	Y.set(&X);

	updateChat();
#ifdef INSTRUMENTATION
	std::chrono::high_resolution_clock::time_point t2 = std::chrono::high_resolution_clock::now();
	timePerBlock.increment(0, 11, (float) std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
	occurencePerBlock.increment(0, 11, 1);
#endif // INSTRUMENTATION

}

void OPFADMMConsGPU::init(const Simparam& sim, const StudyCase& cas)
{

	if (_CoresChatBegin.getPos()) {

		_CoresChatBegin.transferCPU();
		_indiceBusBegin.transferCPU();

		Ancestor.transferCPU();
		PosChild.transferCPU();

		_indiceChildBegin.transferCPU();
		Childs.transferCPU();

		ZsIm.transferCPU();
		ZsRe.transferCPU();
	}


	// intitilisation des matrixs et variables 

	clock_t t = clock();
	
	std::cout << "init " << std::endl;
	_rho = sim.getRho();
	
	_iterG = sim.getIterG();
	_stepG = sim.getStepG();

	_nAgent = cas.getNagent();

	_nBus = cas.getNBus();
	_nBusWLoss = _nBus + 1;
	_nLine = cas.getNLine(true); // ne doit pas être réduit ici !!!

	_debutloss = 3 * _nLine + 5 * _nBus + 2 * (_nAgent - 1); // L = nChild.sum()
	_sizeOPFADMMConsTotal = _debutloss;
	_sizeChat = 4 * _nBus + 2 * _nAgent;

	if (losstype == LossType::CURRENT) {
		_sizeOPFADMMConsTotal += (_nBus + 2); // pertes et courants sauf premier bus ou + 2
	}
	else if (losstype == LossType::POWER) {
		_sizeOPFADMMConsTotal += _nAgent;
	}

	_numBlocksB = ceil((_nBus + _blockSize - 1) / _blockSize);
	_numBlocksH = ceil((_sizeOPFADMMConsTotal + _blockSize - 1) / _blockSize);
	_numBlocksN = ceil((_nAgent + _blockSize - 1) / _blockSize);

	


	//std::cout << _nAgent << " " << _nBus << " " << _nLine << std::endl;
	_CoresAgentBus = MatrixGPU(cas.getCoresAgentBusLin(), 1);
	_CoresAgentBusBegin = MatrixGPU(cas.getCoresAgentBusLinBegin(), 1);
	_nAgentByBus = MatrixGPU(cas.getNagentByBus(), 1);
	_nAgentByBusCPU = cas.getNagentByBus();
	PosAgent = MatrixGPU(_nAgent, 1, 0, 1);

	initPosAgent << <_nBus, _blockSizeSmall >> > (PosAgent._matrixGPU, _nAgentByBus._matrixGPU, _CoresAgentBusBegin._matrixGPU, _CoresAgentBus._matrixGPU);
	
	nChildCPU = MatrixCPU(_nBus, 1);

	CoresLineBusCPU = cas.getCoresLineBus(true);
	CoresLineBus = MatrixGPU(CoresLineBusCPU, 1);

	_CoresBusAgent = MatrixGPU(cas.getCoresBusAgentLin(), 1); // Cores[n] = b

	Ancestor = MatrixGPU(_nBus, 1, 0); // A_i = bus antécédent de i
	PosChild = MatrixGPU(_nBus, 1, 0); // indice du bus i dans Child[Ai]
	Ancestor.set(0, 0, -1); // the slack bus has no ancestor

	ZsRe = cas.getZsRe();
	ZsIm = cas.getZsImag();
	ZsNorm = MatrixCPU(_nLine, 1);

	if (!chekcase()) {
		throw std::invalid_argument("not a radial case");
	}
	if (losstype != LossType::CURRENT && losstype != LossType::POWER) {
		throw std::invalid_argument("unkown lossType");
	}



	for (int lold = 0; lold < _nLine; lold++) {
		int l = lold + 1;
		int busTo = l;
		int busFrom = CoresLineBusCPU.get(lold, 0);
		Ancestor.set(busTo, 0, busFrom);
		nChildCPU.increment(busFrom, 0, 1);
		ZsNorm.set(lold, 0, ZsRe.get(lold, 0) * ZsRe.get(lold, 0) + ZsIm.get(lold, 0) * ZsIm.get(lold, 0));
	}
	nChild = MatrixGPU(nChildCPU, 1);

	/*
	ZsNorm.display();*/


	_rhoInv = 1 / _rho;
	resF = MatrixCPU(3, (_iterG / _stepG) + 1);


	std::cout << " local resolution " << std::endl;
	// local resolution
	tempN2 = MatrixGPU(2 * _nAgent, 1, 0, 1);
	tempB2 = MatrixGPU(2 * _nBus, 1, 0, 1);

	CoresSoloBusAgent = MatrixGPU(_nBus, 1, -1, 1);
	Pn = MatrixGPU(sim.getPn(), 1); // not the real agent
	Pmin = MatrixGPU(cas.getPmin(), 1); 
	Pmax = MatrixGPU(cas.getPmax(), 1); // idem

	// the loss provider
	/*Pmin.set(0, 0, 0);
	Pmax.set(0, 0, 0);
	Pmin.set(_nAgent, 0, 0);
	Pmax.set(_nAgent, 0, 0);*/

	Pbmax = MatrixGPU(2 * _nBus, 1, 0, 1);
	Pbmin = MatrixGPU(2 * _nBus, 1, 0, 1);
	Pb = MatrixGPU(2 * _nBus, 1, 0, 1);
	//Pmin.display();
	//Pn.display();


	Cost1 = MatrixGPU(cas.geta(), 1);
	Cost2 = MatrixGPU(cas.getb(), 1);

	Pn.preallocateReduction();
	if (Pn.max2() < 0.00001) {
		Pn.add(&Pmin, &Pmax);
		Pn.divide(2);
	}
	Pn.set(0, 0, 0, 1);
	Pmin.set(0, 0, 0, 1);
	Pmax.set(0, 0, 0, 1);
	Pn.set(_nAgent, 0, 0, 1);
	Pmin.set(_nAgent, 0, 0, 1);
	Pmax.set(_nAgent, 0, 0, 1);

	_nAgentByBusCPU.increment(0, 0, -1);
	_nAgentOn0 = _nAgentByBusCPU.get(0, 0);

	removeLossAgent << <1, 1 >> > (_nAgentByBus._matrixGPU, _CoresAgentBusBegin._matrixGPU);
	 
	ComputePFromAgentToBus();
	 
	//_nAgentByBus.display();


	std::cout << " creation " << std::endl;
	X = MatrixGPU(_sizeOPFADMMConsTotal, 1, 0, 1);
	Ypre = MatrixGPU(_sizeOPFADMMConsTotal, 1, 0, 1);
	Y = MatrixGPU(_sizeOPFADMMConsTotal, 1, 0, 1);
	Y.preallocateReduction();

	Mu = MatrixGPU(_sizeOPFADMMConsTotal, 1, 0, 1);

	tempNN = MatrixGPU(_nAgent, _nAgent, 0, 1);
	//tempM1 = new MatrixGPU[_nAgent];
	tempM = MatrixGPU(_sizeOPFADMMConsTotal, 1, 0, 1);

	sizeOPFADMMConsGPU = MatrixGPU(_nBusWLoss, 1, 0, 1);
	sizeOPFADMMConsGPU.preallocateReduction();
	sizeOPFADMMConsGPUBig = MatrixGPU(_sizeOPFADMMConsTotal, 1, 0, 1);

	indiceBusBeginCPU = MatrixCPU(_nBusWLoss, 1);
	_indiceBusBeginBig = MatrixGPU(_sizeOPFADMMConsTotal, 1, 0, 1);
	CoresChatBeginCPU = MatrixCPU(_nBusWLoss, 1);


	int debut = 0;
	int debutChat = 0;
	for (int i = 0; i < _nBus; i++) {
		int m = nChildCPU.get(i, 0);
		int nB = _nAgentByBusCPU.get(i, 0);
		indiceBusBeginCPU.set(i, 0, debut);
		CoresChatBeginCPU.set(i, 0, debutChat);
		int sizeA = m * 3 + 5 + 2 * nB;
		debut += sizeA;
		debutChat += 4 + 2 * nB;
	}
	indiceBusBeginCPU.set(_nBus, 0, debut);
	CoresChatBeginCPU.set(_nBus, 0, debutChat);


	_CoresChatBegin = MatrixGPU(CoresChatBeginCPU, 1);
	_indiceBusBegin = MatrixGPU(indiceBusBeginCPU, 1);
	defineSizeBig << <_nBusWLoss, _blockSize >> > (sizeOPFADMMConsGPUBig._matrixGPU, nChild._matrixGPU, _indiceBusBegin._matrixGPU, sizeOPFADMMConsGPU._matrixGPU, _indiceBusBeginBig._matrixGPU, _nAgentByBus._matrixGPU, losstype, _nBus, _nAgent);
	 

	_sizeOPFADMMConsMax = sizeOPFADMMConsGPU.max2();
	Hinv = MatrixGPU(_sizeOPFADMMConsTotal, _sizeOPFADMMConsMax, 0, 1);
	Q = MatrixGPU(_sizeOPFADMMConsTotal, 1, 0, 1);

	Childs = MatrixGPU(_nLine, 1);
	PosChild = MatrixGPU(_nBus, 1, -1);
	_indiceChildBegin = MatrixGPU(_nBus, 1);
	Chat = MatrixGPU(_sizeChat, 1, 0, 1);

	MatrixGPU lowerBound(cas.getLowerBound(), 1); //voltage angle, voltage, line...
	MatrixGPU upperBound(cas.getUpperBound(), 1); //voltage angle, voltage, line...

	VoltageLimit = MatrixGPU(2, _nBus, 0, 1); // min, max
	VoltageLimitCPU = MatrixCPU(2, _nBus, 0);
	VoltageLimitReal = MatrixGPU(2, _nBus, 0, 1); // min, max

	if (cas.isCurrentLimit()) {
		FluxLimit = MatrixGPU(cas.getCurrentLimit(), 1);
		isCurrentLimited = true;
	}
	else {
		FluxLimit = MatrixGPU(_nLine, 1, 1000, 1); // max
	}
	//FluxLimit.display();
	initVoltageBound << < _numBlocksB, _blockSize >> > (VoltageLimitReal._matrixGPU, VoltageLimit._matrixGPU, lowerBound._matrixGPU, upperBound._matrixGPU, nChild._matrixGPU, _nBus);
	VoltageLimit.toMatCPU(VoltageLimitCPU);


	int debutChild = 0;
	MatrixCPU nChildTemp(_nBus, 1, 0);

	for (int i = 0; i < _nBus; i++) {
		if (i > 0) {
			_indiceChildBegin.set(i - 1, 0, debutChild);

			int Ai = Ancestor.get(i, 0);
			Childs.set(_indiceChildBegin.get(Ai, 0) + nChildTemp.get(Ai, 0), 0, i);
			PosChild.set(i, 0, nChildTemp.get(Ai, 0));
			nChildTemp.increment(Ai, 0, 1);
			debutChild += nChildCPU.get(i - 1, 0);
		}

	}
	Ancestor.transferGPU();
	PosChild.transferGPU();

	debut = 0;
	//std::cout << " Hinv " << std::endl;
	for (int i = 0; i < _nBus; i++) {
		// (Pi, Qi, li, vi, pn..., qn..., vai, Pci ..., Qci... , lci...) !!!!!
		int m = nChildCPU.get(i, 0);
		int nB = _nAgentByBusCPU.get(i, 0);
		int sizeA = m * 3 + 5 + 2 * nB;
		MatrixCPU A(2 + 1 * (i > 0), sizeA);

		if (i > 0) {
			A.set(2, 0, 2 * ZsRe.get(i - 1, 0));
			A.set(2, 1, 2 * ZsIm.get(i - 1, 0));
			A.set(2, 2, -ZsNorm.get(i - 1, 0));
			A.set(2, 3, -1);
			A.set(2, 4 + 2 * nB, 1); // vai
			A.set(0, 0, -1);
			A.set(1, 1, -1);
		}
		for (int In = 0; In < nB; In++) {
			A.set(0, 4 + In, 1);
			A.set(1, 4 + nB + In, 1);
		}

		for (int j = 0; j < m; j++) {
			int c = Childs.get(_indiceChildBegin.get(i, 0) + j, 0);
			A.set(0, 5 + 2 * nB + j, 1); // Pci
			A.set(1, 5 + 2 * nB + m + j, 1); // Qci
			A.set(0, 5 + 2 * nB + 2 * m + j, -ZsRe.get(c - 1, 0)); // -R l
			A.set(1, 5 + 2 * nB + 2 * m + j, -ZsIm.get(c - 1, 0)); // -X l
		}

		//A.display();

		MatrixCPU temp33(2 + 1 * (i > 0), 2 + 1 * (i > 0));
		MatrixCPU temp3M(2 + 1 * (i > 0), sizeA);
		MatrixCPU tempMM(sizeA, sizeA);
		MatrixCPU tempMMbis(sizeA, sizeA);

		temp33.multiplyTrans(&A, &A);
		temp33.invertGaussJordan(&temp33);
		temp3M.MultiplyMatMat(&temp33, &A);
		tempMM.multiplyTrans(&A, &temp3M, 0);

		tempMMbis.setEyes(-1);
		tempMMbis.add(&tempMM);
		MatrixGPU tempMMGPU = MatrixGPU(tempMMbis, 1);

		Hinv.setBloc(debut, debut + sizeA, 0, sizeA, &tempMMGPU);
		debut += sizeA;
	}

	// bus factice
	int sizeA = 0;
	MatrixGPU A;
	switch (losstype)
	{
	case LossType::POWER:
		sizeA = 2 * _nAgent;
		A = MatrixGPU(sizeA, sizeA);
		A.setEyes(-1);
		A.set(0, 0, 0);
		A.set(_nAgent, _nAgent, 0);
		for (int i = 1; i < _nAgent; i++) {
			A.set(0, i, 1); // sum(p) + Ploss = 0
			A.set(_nAgent, i + _nAgent, 1); // Qloss + sum(q) = 0
		}
		A.transferGPU();
		Hinv.setBloc(debut, debut + sizeA, 0, sizeA, &A);
		/*for (int i = 0; i < _nAgentTrue; i++) {
			A[_nBus].set(0, i, 1); // sum(p) + Ploss = 0
			A[_nBus].set(1, i + _nAgentTrue, 1); // Qloss + sum(q) = 0
		}*/
		break;
	case LossType::CURRENT:
		sizeA = 2 + _nBus;
		A = MatrixGPU(sizeA, sizeA);
		A.setEyes(-1);
		A.set(0, 0, 0);
		A.set(1, 1, 0);
		for (int i = 0; i < _nLine; i++) {
			A.set(0, i + 3, ZsRe.get(i, 0)); // sum(p) + Ploss = 0
			A.set(1, i + 3, ZsIm.get(i, 0)); // Qloss + sum(q) = 0
		}
		A.transferGPU();
		Hinv.setBloc(debut, debut + sizeA, 0, sizeA, &A);
		//A[_nBus].set(0, 0, 1); // ploss
		//A[_nBus].set(1, 1, 1); // qloss
		break;
	}
	Hinv.divide(_rho);
	ZsRe.transferGPU();
	ZsIm.transferGPU();
	_indiceChildBegin.transferGPU();
	Childs.transferGPU();

	//	std::cout << " init valeur " << std::endl;

	initPQAgentV << < _nBus, _blockSizeSmall >> > (X._matrixGPU, _indiceBusBegin._matrixGPU, _CoresAgentBus._matrixGPU, _nAgentByBus._matrixGPU, _CoresAgentBusBegin._matrixGPU, Pn._matrixGPU, _nAgent);
	

	initDFSPQ << <1, _nBus, _nBus* (8*sizeof(bool) + sizeof(int)) >> > (X._matrixGPU, Pb._matrixGPU, nChild._matrixGPU, Childs._matrixGPU, _indiceBusBegin._matrixGPU, _indiceChildBegin._matrixGPU, _nBus);
	//std::cout << " X " << std::endl;
	//X.display(true);

	communicateX << <_nBusWLoss, _blockSize >> > (X._matrixGPU, nChild._matrixGPU, Ancestor._matrixGPU, Childs._matrixGPU, _indiceBusBegin._matrixGPU, _indiceChildBegin._matrixGPU, _nAgentByBus._matrixGPU, _CoresBusAgent._matrixGPU, PosAgent._matrixGPU, losstype, _nBus, _nAgent);
	 

	computeLoss();
	 

	Y.set(&X);

	// bus factice




	//std::cout << "updateChat" << std::endl;
	updateChat();
	 

	//std::cout << "rho " << _rhog << " rhoL " << _rhol << " rho1 " << _rho1 << std::endl;
	//std::cout << "fin init temps : " << (float)(clock() - t) / CLOCKS_PER_SEC << std::endl;
	//std::cout << "---------------------------------------------------------------------------------------" << std::endl;
}

void OPFADMMConsGPU::solveConsensus(float eps, MatrixGPU* PSO)
{
	float epsG = eps;
	float resG = 2 * epsG;
	_iterGlobal = 0;
	timeOPF = clock();

	/*std::cout << "****** Solve Consensus OPF part ************" << std::endl;
	Chat[1].display();
	Cost2.display();*/

	while ((_iterGlobal < _iterG) && (resG > epsG)) {

		
		updateXWOCurrent();
		CommunicationX();

		
		updateGlobalProb();
		updateMu();
		
		updateChat();

		// FB 4
		if (!(_iterGlobal % _stepG)) {
			resG = updateResRhoFixe(_iterGlobal / _stepG);
		}
		//std::cout << iterGlobal << " " << _iterLocal << " " << resL << " " << resF.get(0, iterGlobal / stepG) << " " << resF.get(1, iterGlobal / stepG) << std::endl;

		_iterGlobal++;
	}
	
	//if (_iterG == _iterGlobal) {
		//std::cout << "OPF GPU : " << _iterGlobal << " " << resF.get(0, (_iterGlobal - 1) / _stepG) << " " << resF.get(1, (_iterGlobal - 1) / _stepG) << " " << resF.get(2, (_iterGlobal - 1) / _stepG) << std::endl;
	//}
	//X[1].display();
	
	//setPnFromX << < _nBus, _blockSizeSmall >> > (Pn._matrixGPU, X._matrixGPU, _indiceBusBegin._matrixGPU, _CoresAgentBus._matrixGPU, _nAgentByBus._matrixGPU, _CoresAgentBusBegin._matrixGPU, _nAgent);

	//MatrixCPU PnCPU;
	//Pn.toMatCPU(PnCPU);
	
	
	//X.display(true);
	//Y[_nBus].display();
	//Pn.display(true);

	//Pn.set(0, 0, _Ploss);
	//Pn.set(_nAgent, 0, _Qloss);
	
	PSO->set(&Pn);
	PSO->set(0, 0, getPLoss(), true);
	PSO->set(_nAgent, 0, getQLoss(), true);

	//PSO->display();

	timeOPF = clock() - timeOPF;

}


void OPFADMMConsGPU::solveConsensus(float eps, MatrixCPU* PSO) {
	
	solveConsensus(eps, &PSOGPU);

	PSOGPU.toMatCPU(*PSO);

}


void OPFADMMConsGPU::initConsensus(const Simparam& sim, const StudyCase& cas, float rhoSO)
{
	if (_CoresChatBegin.getPos()) {

		_CoresChatBegin.transferCPU();
		_indiceBusBegin.transferCPU();

		Ancestor.transferCPU();
		PosChild.transferCPU();

		_indiceChildBegin.transferCPU();
		Childs.transferCPU();

		ZsIm.transferCPU();
		ZsRe.transferCPU();
	}
	// intitilisation des matrixs et variables 

	clock_t t = clock();
	//std::cout << "init OPF " << std::endl;
	_rho = sim.getRho();
	_rhoSO = rhoSO;

	_iterG = sim.getIterG();
	_stepG = sim.getStepG();

	_nAgent = cas.getNagent();

	_nBus = cas.getNBus();
	_nBusWLoss = _nBus + 1;
	_nLine = cas.getNLine(true); // ne doit pas être réduit ici !!!

	_debutloss = 3 * _nLine + 5 * _nBus + 2 * (_nAgent - 1); // L = nChild.sum()
	_sizeOPFADMMConsTotal = _debutloss;
	_sizeChat = 4 * _nBus + 2 * _nAgent;

	if (losstype == LossType::CURRENT) {
		_sizeOPFADMMConsTotal += (_nBus + 2); // pertes et courants sauf premier bus ou + 2
	}
	else if (losstype == LossType::POWER) {
		_sizeOPFADMMConsTotal += _nAgent;
	}

	_numBlocksB = ceil((_nBus + _blockSize - 1) / _blockSize);
	_numBlocksH = ceil((_sizeOPFADMMConsTotal + _blockSize - 1) / _blockSize);
	_numBlocksN = ceil((_nAgent + _blockSize - 1) / _blockSize);
	
	
	//std::cout << _nAgent << " " << _nBus << " " << _nLine << std::endl;

	_CoresAgentBus = MatrixGPU(cas.getCoresAgentBusLin(), 1);
	_CoresAgentBusBegin = MatrixGPU(cas.getCoresAgentBusLinBegin(), 1);
	_nAgentByBus = MatrixGPU(cas.getNagentByBus(), 1);
	_nAgentByBusCPU = cas.getNagentByBus();
	PosAgent = MatrixGPU(_nAgent, 1, 0, 1);

	////CHECK_LAST_CUDA_ERROR();
	initPosAgent << <_nBus, _blockSizeSmall >> > (PosAgent._matrixGPU, _nAgentByBus._matrixGPU, _CoresAgentBusBegin._matrixGPU, _CoresAgentBus._matrixGPU);
	////CHECK_LAST_CUDA_ERROR();


	nChildCPU = MatrixCPU(_nBus, 1);

	CoresLineBusCPU = cas.getCoresLineBus(true);
	CoresLineBus = MatrixGPU(CoresLineBusCPU, 1);

	_CoresBusAgent = MatrixGPU(cas.getCoresBusAgentLin(), 1); // Cores[n] = b

	Ancestor = MatrixGPU(_nBus, 1, 0); // A_i = bus antécédent de i
	PosChild = MatrixGPU(_nBus, 1, 0); // indice du bus i dans Child[Ai]
	Ancestor.set(0, 0, -1); // the slack bus has no ancestor
	
	ZsRe = cas.getZsRe();
	ZsIm = cas.getZsImag();	
	ZsNorm = MatrixCPU(_nLine, 1);

	if (!chekcase()) {
		throw std::invalid_argument("not a radial case");
	}
	if (losstype != LossType::CURRENT && losstype != LossType::POWER ) {
		throw std::invalid_argument("unkown lossType");
	}
	

	
	for (int lold = 0; lold < _nLine; lold++) {
		int l = lold + 1;
		int busTo = l;
		int busFrom = CoresLineBusCPU.get(lold, 0);
		Ancestor.set(busTo, 0, busFrom);
		nChildCPU.increment(busFrom, 0, 1);
		ZsNorm.set(lold, 0, ZsRe.get(lold, 0) * ZsRe.get(lold, 0) + ZsIm.get(lold, 0) * ZsIm.get(lold, 0));
	}
	nChild = MatrixGPU(nChildCPU, 1);

	/*
	ZsNorm.display();*/


	_rhoInv = 1 / _rho;
	resF = MatrixCPU(3, (_iterG / _stepG) + 1);


	//std::cout << " local resolution " << std::endl;
	// local resolution
	tempN2 = MatrixGPU(2 * _nAgent, 1, 0, 1);
	tempB2 = MatrixGPU(2 * _nBus, 1, 0, 1);

	CoresSoloBusAgent = MatrixGPU(_nBus, 1, -1, 1);

	Pn = MatrixGPU(sim.getPn(), 1); // not the real agent
	PSOGPU = MatrixGPU(2 * _nAgent, 1, 0, 1);
	Pmin = MatrixGPU(2 * _nAgent, 1, -1000000, 1); // must not be the real one
	Pmax = MatrixGPU(2 * _nAgent, 1, 1000000, 1); // idem

	// the loss provider
	/*Pmin.set(0, 0, 0);
	Pmax.set(0, 0, 0);
	Pmin.set(_nAgent, 0, 0);
	Pmax.set(_nAgent, 0, 0);*/

	Pbmax = MatrixGPU(2 * _nBus, 1, 0, 1);
	Pbmin = MatrixGPU(2 * _nBus, 1, 0, 1);
	Pb = MatrixGPU(2 * _nBus, 1, 0, 1);
	//Pmin.display();
	//Pn.display();


	Cost1 = MatrixGPU(2 * _nAgent, 1, _rhoSO, 1); //
	Cost1.set(0, 0, 0, true);
	Cost1.set(_nAgent, 0, 0, true);
	Cost2 = MatrixGPU(2 * _nAgent, 1, 0, 1);
	etaSO = MatrixGPU(2 * _nAgent, 1, 0, 1);

	_nAgentByBusCPU.increment(0, 0, -1);
	_nAgentOn0 = _nAgentByBusCPU.get(0, 0);

	////CHECK_LAST_CUDA_ERROR();
	removeLossAgent << <1, 1 >> > (_nAgentByBus._matrixGPU, _CoresAgentBusBegin._matrixGPU);

	////CHECK_LAST_CUDA_ERROR();
	ComputePFromAgentToBus();
	////CHECK_LAST_CUDA_ERROR();
	//_nAgentByBus.display();


	//std::cout << " creation " << std::endl;
	X = MatrixGPU(_sizeOPFADMMConsTotal, 1, 0, 1);
	Ypre = MatrixGPU(_sizeOPFADMMConsTotal, 1, 0, 1);
	Y = MatrixGPU(_sizeOPFADMMConsTotal, 1, 0, 1);
	Y.preallocateReduction();

	Mu = MatrixGPU(_sizeOPFADMMConsTotal, 1, 0, 1);

	tempNN = MatrixGPU(_nAgent, _nAgent, 0, 1);
	//tempM1 = new MatrixGPU[_nAgent];
	tempM = MatrixGPU(_sizeOPFADMMConsTotal, 1, 0, 1);

	sizeOPFADMMConsGPU = MatrixGPU(_nBusWLoss, 1, 0, 1);
	sizeOPFADMMConsGPU.preallocateReduction();
	sizeOPFADMMConsGPUBig = MatrixGPU(_sizeOPFADMMConsTotal, 1, 0, 1);

	_indiceBusBegin = MatrixGPU(_nBusWLoss, 1);
	_indiceBusBeginBig = MatrixGPU(_sizeOPFADMMConsTotal, 1, 0, 1);
	_CoresChatBegin = MatrixGPU(_nBusWLoss, 1);


	int debut = 0;
	int debutChat = 0;
	for (int i = 0; i < _nBus; i++) {
		int m = nChildCPU.get(i, 0);
		int nB = _nAgentByBusCPU.get(i, 0);
		_indiceBusBegin.set(i, 0, debut);
		_CoresChatBegin.set(i, 0, debutChat);
		int sizeA = m * 3 + 5 + 2 * nB;
		debut += sizeA;
		debutChat += 4 + 2 * nB;
	}
	_indiceBusBegin.set(_nBus, 0, debut);
	_CoresChatBegin.set(_nBus, 0, debutChat);


	_CoresChatBegin.transferGPU();
	_indiceBusBegin.transferGPU();
	////CHECK_LAST_CUDA_ERROR();
	defineSizeBig << <_nBusWLoss, _blockSize >> > (sizeOPFADMMConsGPUBig._matrixGPU, nChild._matrixGPU, _indiceBusBegin._matrixGPU, sizeOPFADMMConsGPU._matrixGPU, _indiceBusBeginBig._matrixGPU, _nAgentByBus._matrixGPU, losstype, _nBus, _nAgent);
	////CHECK_LAST_CUDA_ERROR();

	_sizeOPFADMMConsMax = sizeOPFADMMConsGPU.max2();
	Hinv = MatrixGPU(_sizeOPFADMMConsTotal, _sizeOPFADMMConsMax, 0, 1);
	Q = MatrixGPU(_sizeOPFADMMConsTotal, 1, 0, 1);

	Childs = MatrixGPU(_nLine, 1);
	PosChild = MatrixGPU(_nBus, 1, -1);
	_indiceChildBegin = MatrixGPU(_nBus, 1);

	Chat = MatrixGPU(_sizeChat, 1, 0, 1);

	MatrixGPU lowerBound(cas.getLowerBound(), 1); //voltage angle, voltage, line...
	MatrixGPU upperBound(cas.getUpperBound(), 1); //voltage angle, voltage, line...

	VoltageLimit = MatrixGPU(2, _nBus, 0, 1); // min, max
	VoltageLimitReal = MatrixGPU(2, _nBus, 0, 1); // min, max


	if (cas.isCurrentLimit()) {
		FluxLimit = MatrixGPU(cas.getCurrentLimit(), 1);
		isCurrentLimited = true;
	}
	else {
		FluxLimit = MatrixGPU(_nLine, 1, 1000, 1); // max
	}
	//FluxLimit.display();
	////CHECK_LAST_CUDA_ERROR();
	initVoltageBound << < _numBlocksB, _blockSize >> > (VoltageLimitReal._matrixGPU, VoltageLimit._matrixGPU, lowerBound._matrixGPU, upperBound._matrixGPU, nChild._matrixGPU, _nBus);
	////CHECK_LAST_CUDA_ERROR();

	int debutChild = 0;
	MatrixCPU nChildTemp(_nBus, 1, 0);
	

	for (int i = 0; i < _nBus; i++) {
		if (i > 0) {
			_indiceChildBegin.set(i - 1, 0, debutChild);

			int Ai = Ancestor.get(i, 0);
			Childs.set(_indiceChildBegin.get(Ai, 0) + nChildTemp.get(Ai, 0), 0, i);
			PosChild.set(i, 0, nChildTemp.get(Ai, 0));
			nChildTemp.increment(Ai, 0, 1);
			debutChild += nChildCPU.get(i - 1, 0);
		}

	}
	Ancestor.transferGPU();
	PosChild.transferGPU();

	debut = 0;
	for (int i = 0; i < _nBus; i++) {
		// (Pi, Qi, li, vi, pn..., qn..., vai, Pci ..., Qci... , lci...) !!!!!
		int m = nChildCPU.get(i, 0);
		int nB = _nAgentByBusCPU.get(i, 0);
		int sizeA = m * 3 + 5 + 2 * nB;
		MatrixCPU A(2 + 1 * (i > 0), sizeA);

		if (i > 0) {
			A.set(2, 0, 2 * ZsRe.get(i - 1, 0));
			A.set(2, 1, 2 * ZsIm.get(i - 1, 0));
			A.set(2, 2, -ZsNorm.get(i - 1, 0));
			A.set(2, 3, -1);
			A.set(2, 4 + 2 * nB, 1); // vai
			A.set(0, 0, -1);
			A.set(1, 1, -1);
		}
		for (int In = 0; In < nB; In++) {
			A.set(0, 4 + In, 1);
			A.set(1, 4 + nB + In, 1);
		}

		for (int j = 0; j < m; j++) {
			int c = Childs.get(_indiceChildBegin.get(i, 0) + j, 0);
			A.set(0, 5 + 2 * nB + j, 1); // Pci
			A.set(1, 5 + 2 * nB + m + j, 1); // Qci
			A.set(0, 5 + 2 * nB + 2 * m + j, -ZsRe.get(c - 1, 0)); // -R l
			A.set(1, 5 + 2 * nB + 2 * m + j, -ZsIm.get(c - 1, 0)); // -X l
		}

		//A.display();

		MatrixCPU temp33(2 + 1 * (i > 0), 2 + 1 * (i > 0));
		MatrixCPU temp3M(2 + 1 * (i > 0), sizeA);
		MatrixCPU tempMM(sizeA, sizeA);
		MatrixCPU tempMMbis(sizeA, sizeA);

		temp33.multiplyTrans(&A, &A);
		temp33.invertGaussJordan(&temp33);
		temp3M.MultiplyMatMat(&temp33, &A);
		tempMM.multiplyTrans(&A, &temp3M, 0);

		tempMMbis.setEyes(-1);
		tempMMbis.add(&tempMM);
		MatrixGPU tempMMGPU = MatrixGPU(tempMMbis, 1);

		Hinv.setBloc(debut, debut + sizeA, 0, sizeA, &tempMMGPU);
		debut += sizeA;
	}

	// bus factice
	int sizeA = 0;
	MatrixGPU A;
	switch (losstype)
	{
	case LossType::POWER:
		sizeA = 2 *_nAgent;
		A = MatrixGPU(sizeA, sizeA);
		A.setEyes(-1);
		A.set(0, 0, 0);
		A.set(_nAgent, _nAgent, 0);
		for (int i = 1; i < _nAgent; i++) {
			A.set(0, i, 1); // sum(p) + Ploss = 0
			A.set(_nAgent, i + _nAgent, 1); // Qloss + sum(q) = 0
		}
		A.transferGPU();
		Hinv.setBloc(debut, debut + sizeA, 0, sizeA, &A);
		/*for (int i = 0; i < _nAgentTrue; i++) {
			A[_nBus].set(0, i, 1); // sum(p) + Ploss = 0
			A[_nBus].set(1, i + _nAgentTrue, 1); // Qloss + sum(q) = 0
		}*/
		break;
	case LossType::CURRENT:
		sizeA = 2 + _nBus;
		A = MatrixGPU(sizeA, sizeA);
		A.setEyes(-1);
		A.set(0, 0, 0);
		A.set(1, 1, 0);
		for (int i = 0; i < _nLine; i++) {
			A.set(0, i + 3, ZsRe.get(i, 0)); // sum(p) + Ploss = 0
			A.set(1, i + 3, ZsIm.get(i, 0)); // Qloss + sum(q) = 0
		}
		A.transferGPU();
		Hinv.setBloc(debut, debut + sizeA, 0, sizeA, &A);
		//A[_nBus].set(0, 0, 1); // ploss
		//A[_nBus].set(1, 1, 1); // qloss
		break;
	}

	Hinv.divide(_rho);

	ZsRe.transferGPU();
	ZsIm.transferGPU();
	_indiceChildBegin.transferGPU();
	Childs.transferGPU();


	//std::cout << " init valeur " << std::endl;
	////CHECK_LAST_CUDA_ERROR();
	initPQAgentV << < _nBus, _blockSizeSmall >> > (X._matrixGPU, _indiceBusBegin._matrixGPU, _CoresAgentBus._matrixGPU, _nAgentByBus._matrixGPU, _CoresAgentBusBegin._matrixGPU, Pn._matrixGPU, _nAgent);
	////CHECK_LAST_CUDA_ERROR();

	////CHECK_LAST_CUDA_ERROR();
	initDFSPQ << <1, _nBus, _nBus* (8 * sizeof(bool) + sizeof(int)) >> > (X._matrixGPU, Pb._matrixGPU, nChild._matrixGPU, Childs._matrixGPU, _indiceBusBegin._matrixGPU, _indiceChildBegin._matrixGPU, _nBus);
	////CHECK_LAST_CUDA_ERROR();

	communicateX << <_nBusWLoss, _blockSize >> > (X._matrixGPU, nChild._matrixGPU, Ancestor._matrixGPU, Childs._matrixGPU, _indiceBusBegin._matrixGPU, _indiceChildBegin._matrixGPU, _nAgentByBus._matrixGPU, _CoresBusAgent._matrixGPU, PosAgent._matrixGPU, losstype, _nBus, _nAgent);
	////CHECK_LAST_CUDA_ERROR();

	computeLoss();
	////CHECK_LAST_CUDA_ERROR();

	Y.set(&X);

	// bus factice

	
	

	//std::cout << "updateChat" << std::endl;
	updateChat();
	
	hipDeviceSynchronize();
	//CHECK_LAST_CUDA_ERROR();
	//std::cout << "rho " << _rhog << " rhoL " << _rhol << " rho1 " << _rho1 << std::endl;
	//std::cout << "fin init temps : " << (float)(clock() - t) / CLOCKS_PER_SEC << std::endl;
	//std::cout << "---------------------------------------------------------------------------------------" << std::endl;
}


void OPFADMMConsGPU::updateConsensus(MatrixGPU* Pmarket)
{
	
	//Pmarket->display(true);
	CHECK_LAST_CUDA_ERROR();
	updateConsensusGPU << <_numBlocksN, _blockSize >> > (Cost2._matrixGPU, etaSO._matrixGPU, Pn._matrixGPU, Pmarket->_matrixGPU, _rhoSO, _nAgent);
	
	CHECK_LAST_CUDA_ERROR();
	/*std::cout << "Cost 2" << std::endl;
	Cost2.display(true);
	std::cout << "*********" << std::endl;*/

}

void OPFADMMConsGPU::updateConsensus(MatrixCPU* Pmarket) {

	PSOGPU.set(Pmarket);
	updateConsensus(&PSOGPU);

}



void OPFADMMConsGPU::updateGlobalProb() {
	
	
	Ypre.swap(&Y);
	int numBlock = _sizeOPFADMMConsTotal;
	switch (_blockSize) {
	case 512:
		updateY<512> << <numBlock, _blockSize >> > (Y._matrixGPU, Hinv._matrixGPU, Q._matrixGPU, sizeOPFADMMConsGPUBig._matrixGPU, _indiceBusBeginBig._matrixGPU, _sizeOPFADMMConsMax);
		break;
	case 256:
		updateY<256> << <numBlock, _blockSize >> > (Y._matrixGPU, Hinv._matrixGPU, Q._matrixGPU, sizeOPFADMMConsGPUBig._matrixGPU, _indiceBusBeginBig._matrixGPU, _sizeOPFADMMConsMax);
		break;
	case 128:
		updateY<128> << <numBlock, _blockSize >> > (Y._matrixGPU, Hinv._matrixGPU, Q._matrixGPU, sizeOPFADMMConsGPUBig._matrixGPU, _indiceBusBeginBig._matrixGPU, _sizeOPFADMMConsMax);
		break;
	case 64:
		updateY< 64> << <numBlock, _blockSize >> > (Y._matrixGPU, Hinv._matrixGPU, Q._matrixGPU, sizeOPFADMMConsGPUBig._matrixGPU, _indiceBusBeginBig._matrixGPU, _sizeOPFADMMConsMax);
		break;
	case 32:
		updateY< 32> << <numBlock, _blockSize >> > (Y._matrixGPU, Hinv._matrixGPU, Q._matrixGPU, sizeOPFADMMConsGPUBig._matrixGPU, _indiceBusBeginBig._matrixGPU, _sizeOPFADMMConsMax);
		break;
	case 16:
		updateY< 16> << <numBlock, _blockSize >> > (Y._matrixGPU, Hinv._matrixGPU, Q._matrixGPU, sizeOPFADMMConsGPUBig._matrixGPU, _indiceBusBeginBig._matrixGPU, _sizeOPFADMMConsMax);
		break;
	case  8:
		updateY<  8> << <numBlock, _blockSize >> > (Y._matrixGPU, Hinv._matrixGPU, Q._matrixGPU, sizeOPFADMMConsGPUBig._matrixGPU, _indiceBusBeginBig._matrixGPU, _sizeOPFADMMConsMax);
		break;
	case  4:
		updateY<  4> << <numBlock, _blockSize >> > (Y._matrixGPU, Hinv._matrixGPU, Q._matrixGPU, sizeOPFADMMConsGPUBig._matrixGPU, _indiceBusBeginBig._matrixGPU, _sizeOPFADMMConsMax);
		break;
	case  2:
		updateY<  2> << <numBlock, _blockSize >> > (Y._matrixGPU, Hinv._matrixGPU, Q._matrixGPU, sizeOPFADMMConsGPUBig._matrixGPU, _indiceBusBeginBig._matrixGPU, _sizeOPFADMMConsMax);
		break;
	case  1:
		updateY<  1> << <numBlock, _blockSize >> > (Y._matrixGPU, Hinv._matrixGPU, Q._matrixGPU, sizeOPFADMMConsGPUBig._matrixGPU, _indiceBusBeginBig._matrixGPU, _sizeOPFADMMConsMax);
		break;
	}

	Y.set(3, 0, 1, 1);
	Y.set(4 + 2 * _nAgentOn0, 0, 1, 1);

	// communication of y, mu

}

void OPFADMMConsGPU::updateX()
{
	/*double x1, x2, x3, x4, c1, c2, c3, c4, lambdaLo, lambdaUp, delta, x3min, x3max, x4max, gamma, k2;
	double c1122;
	int nSol = 0;
	int typeSol = 0;
	int BestRoot = 0;
	double bestGamma = -1;
	double p = 0;
	int nRoot = 0;
	bool neg = false;


	for (int i = 0; i < _nBus; i++) {

		bool goodSol = false;
		k2 = sqrt(2.0 / (nChild.get(i, 0) + 1));
		typeSol = 0;
		if (i == 0) { // slack bus
			goodSol = true;
			c3 = -2 * Chat[i].get(2, 0) / k2;
			x1 = 0;
			x2 = 0;
			x4 = 0;
			x4max = 0;
			x3 = 1 / k2;
			gamma = 0;
			typeSol = 1;
		}
		else {
			c1 = -2 * Chat[i].get(0, 0);
			c2 = -2 * Chat[i].get(1, 0);
			c3 = -2 * Chat[i].get(2, 0) / k2;
			c4 = -2 * Chat[i].get(3, 0);
			c1122 = c1 * c1 + c2 * c2;


			x3min = VoltageLimit.get(i, 0);
			x3max = VoltageLimit.get(i, 1);
			x4max = FluxLimit.get(i - 1, 0);

			// case without constraint

			x1 = -c1 / 2;
			x2 = -c2 / 2;
			x3 = -c3 / 2;
			x4 = -c4 / 2;
			lambdaUp = 0;
			lambdaLo = 0;

			if (x3 < x3min) {
				x3 = x3min;
				lambdaLo = (2 * x3 + c3);
			}
			else if (x3 > x3max) {
				x3 = x3max;
				lambdaUp = -(2 * x3 + c3);
			}

			if (x4 > x4max) {
				x4 = x4max;
			}

			gamma = k2 * x4 - (x1 * x1 + x2 * x2) / x3; // ce n'est pas vraiment gamma, doit être positif
			//std::cout << "x 1 : " << x1 << " " << x2 << " " << x3 * k2 << " " << x4 << " " << (x1 * x1 + x2 * x2) / x3  - k2 * x4 << std::endl;

			if (gamma >= 0) {
				// the solution is good !
				typeSol = 1;
				goodSol = true;
			}
			else {
				if (c1122 == 0) { // cas dégénéré
					std::cout << " bus " << i << " : c1= " << c1 << " c2=" << c2 << " c4=" << c4 << " gamma= " << gamma << std::endl;
					x4 = 0;
					goodSol = true;
				}
				else if (gamma > bestGamma) {
					typeSol = 1;
					bestGamma = gamma;
				}
			}
		}

		// cas x4 = x4 max 

			//x3 = x3max
		if (!goodSol) {
			x4 = x4max;
			x3 = x3max;
			p = sqrt((k2 * x4) / (c1122 * x3)); // plus ou mois ce truc !!!
			x1 = p * c1 * x3;
			x2 = p * c2 * x3;
			if (abs(c1) > 0) {
				gamma = -(2 * x1 + c1) * x3 / (2 * x1);
			}
			else {
				gamma = -(2 * x2 + c2) * x3 / (2 * x2);
			}
			lambdaUp = -(2 * x3 + c3 - gamma * (x1 * x1 + x2 * x2) / (x3 * x3));
			delta = k2 * gamma - 2 * x4 - c4;
			if (gamma >= 0 && lambdaUp >= 0 && delta > 0) {
				typeSol = 5;
				goodSol = true;
			}
			else if (gamma > bestGamma && lambdaUp > bestGamma && delta > bestGamma) {
				typeSol = 5;
				bestGamma = min(min(gamma, lambdaUp), delta);
			}
		}
		if (!goodSol) {
			p = -p;
			x1 = p * c1 * x3;
			x2 = p * c2 * x3;
			if (abs(c1) > 0) {
				gamma = -(2 * x1 + c1) * x3 / (2 * x1);
			}
			else {
				gamma = -(2 * x2 + c2) * x3 / (2 * x2);
			}
			lambdaUp = -(2 * x3 + c3 - gamma * (x1 * x1 + x2 * x2) / (x3 * x3));
			delta = k2 * gamma - 2 * x4 - c4;
			if (gamma >= 0 && lambdaUp >= 0 && delta >= 0) {
				// the solution is good 
				goodSol = true;
				typeSol = 5;
				//nSol = n;
			}
			if (gamma > bestGamma && lambdaUp > bestGamma && delta > bestGamma) {
				typeSol = 5;
				neg = true;
				bestGamma = min(min(gamma, lambdaUp), delta);
			}
		}
		//x3 = x3min
		if (!goodSol) {
			// cas x3 = xmin
			x3 = x3min;
			p = sqrt((k2 * x4) / (c1122 * x3));
			x1 = p * c1 * x3;
			x2 = p * c2 * x3;
			if (abs(c1) > 0) {
				gamma = -(2 * x1 + c1) * x3 / (2 * x1);
			}
			else {
				gamma = -(2 * x2 + c2) * x3 / (2 * x2);
			}

			lambdaLo = -(2 * x3 + c3 - gamma * (x1 * x1 + x2 * x2) / (x3 * x3));
			delta = k2 * gamma - 2 * x4 - c4;
			if (gamma >= 0 && lambdaLo >= 0 && delta >= 0) {
				// the solution is good 
				typeSol = 6;
				goodSol = true;
				//nSol = n;

			}
			else if (gamma > bestGamma && lambdaLo > bestGamma && delta > bestGamma) {
				typeSol = 6;
				bestGamma = min(min(gamma, lambdaLo), delta);
			}
		}
		if (!goodSol) {
			p = -p;
			x1 = p * c1 * x3;
			x2 = p * c2 * x3;
			if (abs(c1) > 0) {
				gamma = -(2 * x1 + c1) * x3 / (2 * x1);
			}
			else {
				gamma = -(2 * x2 + c2) * x3 / (2 * x2);
			}

			lambdaLo = -(2 * x3 + c3 - gamma * (x1 * x1 + x2 * x2) / (x3 * x3));
			delta = k2 * gamma - 2 * x4 - c4;
			if (gamma >= 0 && lambdaLo >= 0 && delta >= 0) {
				// the solution is good 
				goodSol = true;
				typeSol = 6;
				//nSol = n;

			}
			else if (gamma > bestGamma && lambdaLo > bestGamma && delta > bestGamma) {
				typeSol = 6;
				bestGamma = min(min(gamma, lambdaLo), delta);
				neg = true;
			}
		}
		// x3min <x3 < x3max
		if (!goodSol) {
			// cas tension libre
			coefPoly2[0] = (c3 + k2 * x4) / 2;
			coefPoly2[1] = sqrt(k2 * x4 * c1122) / 4;
			//std::cout << " polynome " << coefPoly2[0] << " " << coefPoly2[1] << std::endl;

			nRoot = resolveRealPolynome3without2term(root5, coefPoly2);
			for (int n = 0; n < nRoot; n++) {
				double sqrtX3 = root5[n];
				//std::cout << "root5 " << root5[n] << std::endl;
				if (sqrtX3 >= 0) {
					x3 = sqrtX3 * sqrtX3;
					p = sqrt((k2 * x4) / (c1122 * x3));

					x1 = p * c1 * x3;
					x2 = p * c2 * x3;


					if (abs(c1) > 0) {
						gamma = -(2 * x1 + c1) * x3 / (2 * x1);
					}
					else {
						gamma = -(2 * x2 + c2) * x3 / (2 * x2);
					}
					delta = k2 * gamma - 2 * x4 - c4;
					//std::cout << "x : " << x1 << " " << x2 << " " << x3 * k2 << " " << x4 << " " << gamma << " " << delta << std::endl;
					if (gamma >= 0 && delta >= 0 && x3 <= x3max && x3 >= x3min) {
						// the solution is good 
						typeSol = 7;
						goodSol = true;
						//nSol = n;
						break;
					}
					if (gamma > bestGamma && delta > bestGamma && (x3max - x3) > bestGamma && (x3 - x3min) > bestGamma) {
						typeSol = 7;
						bestGamma = min(min(min(gamma, (x3max - x3)), (x3 - x3min)), delta);
						BestRoot = n;
					}
				}

			}
		}
		if (!goodSol) {
			coefPoly2[0] = (c3 + k2 * x4) / 2;
			coefPoly2[1] = -sqrt(k2 * x4 * c1122) / 4;
			//std::cout << " polynome " << coefPoly2[0] << " " << coefPoly2[1] << std::endl;

			nRoot = resolveRealPolynome3without2term(root6, coefPoly2);
			for (int n = 0; n < nRoot; n++) {
				double sqrtX3 = root6[n];
				//std::cout << "root6 " << root6[n] << std::endl;
				if (sqrtX3 > 0) {
					x3 = sqrtX3 * sqrtX3;
					p = -sqrt((k2 * x4) / (c1122 * x3));

					x1 = p * c1 * x3;
					x2 = p * c2 * x3;

					if (abs(c1) > 0) {
						gamma = -(2 * x1 + c1) * x3 / (2 * x1);
					}
					else {
						gamma = -(2 * x2 + c2) * x3 / (2 * x2);
					}
					delta = k2 * gamma - 2 * x4 - c4;
					//std::cout << "x : " << x1 << " " << x2 << " " << x3 * k2 << " " << x4 << " " << gamma << " " << delta << std::endl;
					if (gamma >= 0 && delta >= 0 && x3 <= x3max && x3 >= x3min) {
						// the solution is good 
						typeSol = 8;
						goodSol = true;
						//nSol = n;
						break;
					}
					if (gamma > bestGamma && delta > bestGamma && (x3max - x3) > bestGamma && (x3 - x3min) > bestGamma) {
						typeSol = 8;
						bestGamma = min(min(min(gamma, (x3max - x3)), (x3 - x3min)), delta);
						BestRoot = n;
						neg = true;
					}
				}

			}

		}
		

		// cas x4 < x4 max 
			// case x3 = x3max lambdaLo = 0 delta = 0
		if (!goodSol) {
			x3 = x3max;

			coefPoly2[0] = 2 * (c4 / (k2 * x3) + 1);
			coefPoly2[1] = 1 / x3;
			coefPoly2[0] = coefPoly2[0] * k2 * k2 / (4 * c1122);
			coefPoly2[1] = coefPoly2[1] * k2 * k2 / (4 * c1122);

			nRoot = resolveRealPolynome3without2term(root2, coefPoly2);

			for (int n = 0; n < nRoot; n++) {
				p = root2[n];

				x1 = p * c1 * x3;
				x2 = p * c2 * x3;
				x4 = (x1 * x1 + x2 * x2) / (x3 * k2);
				gamma = (2 * x4 + c4) / k2;
				lambdaUp = -(2 * x3 + c3 - gamma * (x1 * x1 + x2 * x2) / (x3 * x3));
				//std::cout << "x2 : " << x1 << " " << x2 << " " << x3 * k2 << " " << x4 << " " << gamma << " " << lambdaUp << std::endl;
				if (gamma >= 0 && lambdaUp >= 0 && x4 <= x4max) {
					// the solution is good 
					goodSol = true;
					typeSol = 2;
					//nSol = n;
					break;
				}
				if (gamma > bestGamma && lambdaUp > bestGamma && (x4max - x4) > bestGamma) {
					typeSol = 2;
					bestGamma = min((x4max - x4), min(gamma, lambdaUp));
					BestRoot = n;
				}

			}
		}
		// case x3 = x3min lambdaUp = 0
		if (!goodSol) {
			x3 = x3min;

			coefPoly2[0] = 2 * (c4 / (k2 * x3) + 1);
			coefPoly2[1] = 1 / x3;
			coefPoly2[0] = coefPoly2[0] * k2 * k2 / (4 * c1122);
			coefPoly2[1] = coefPoly2[1] * k2 * k2 / (4 * c1122);

			nRoot = resolveRealPolynome3without2term(root3, coefPoly2);

			for (int n = 0; n < nRoot; n++) {
				p = root3[n];
				//std::cout << "poly " << coefPoly2[0] * p + coefPoly2[1] + p * p * p << std::endl;
				x1 = p * c1 * x3;
				x2 = p * c2 * x3;
				x4 = (x1 * x1 + x2 * x2) / (x3 * k2);
				gamma = (2 * x4 + c4) / k2;
				lambdaLo = (2 * x3 + c3 - gamma * (x1 * x1 + x2 * x2) / (x3 * x3));

				//std::cout << "x 3: " << x1 << " " << x2 << " " << x3 * k2 << " " << x4 << " " << gamma << " " << lambdaLo << std::endl;

				if (gamma >= 0 && lambdaLo >= 0 && x4 <= x4max) {
					// the solution is good !
					typeSol = 3;
					goodSol = true;
					break;
				}
				if (gamma > bestGamma && lambdaLo > bestGamma && (x4max - x4) > bestGamma) {
					typeSol = 3;
					bestGamma = min((x4max - x4), min(gamma, lambdaLo));
					BestRoot = n;
				}
			}
		}
		// case xmin<x3<xmax lambdaLo = 0 lambdaUp = 0
		if (!goodSol) {

			coefPoly3[0] = c1122 / k2 * (2 * c3 / k2 - c4);
			coefPoly3[1] = (c3 - 2 * c4 / k2);
			coefPoly3[2] = -1;
			coefPoly3[0] = coefPoly3[0] * k2 * k2 / (c1122 * c1122);
			coefPoly3[1] = coefPoly3[1] * k2 * k2 / (c1122 * c1122);
			coefPoly3[2] = coefPoly3[2] * k2 * k2 / (c1122 * c1122);

			nRoot = resvolveRealPolynome4without2term(root4, coefPoly3);

			for (int n = 0; n < nRoot; n++) {
				p = root4[n];
				//std::cout << "poly " <<p * p * p * p + coefPoly3[0] * p*p*p + coefPoly3[1]*p + coefPoly3[2] << std::endl;
				x3 = -(c1122 * p + 2 * c3) / (2 * (c1122 * p * p + 2));
				x1 = p * c1 * x3;
				x2 = p * c2 * x3;
				x4 = (x1 * x1 + x2 * x2) / (x3 * k2);
				gamma = (2 * x4 + c4) / k2;
				//std::cout << "x 4: " << x1 << " " << x2 << " " << x3 * k2 << " " << x4 << " " << gamma << " " << std::endl;

				if (gamma >= 0 && x3 <= x3max && x3 >= x3min && x4 <= x4max) {
					// the solution is good !
					typeSol = 4;
					goodSol = true;
					break;
				}if (gamma > bestGamma && (x3max - x3) > bestGamma && (x3 - x3min) > bestGamma && (x4max - x4) > bestGamma) {
					typeSol = 4;
					bestGamma = min((x4max - x4), min(min(gamma, (x3max - x3)), (x3 - x3min)));
					BestRoot = n;
				}
			}
		}



		if (!goodSol) {
			std::cout << "*|*" << bestGamma << " " << typeSol << std::endl;	
			Chat[1].display();
			std::cout << "x : " << x1 << " " << x2 << " " << x3 * k2 << " " << x4 << " " << std::endl;
			if (typeSol == 1) {
				// case without constraint
				x1 = -c1 / 2;
				x2 = -c2 / 2;
				x3 = -c3 / 2;
				x3 = (x3max - x3) * (x3 > x3max) + (x3min - x3) * (x3min > x3) + x3;
				x4 = -c4 / 2; // ou  (x1 * x1 + x2 * x2) / (k2 * x3)
				x4 = (x4max - x4) * (x4 > x4max) + x4;
			}
			else if (typeSol > 4) {
				x4 = x4max;
				if (typeSol == 5) {
					x3 = x3max;
				}
				else if (typeSol == 6) {
					x3 = x3min;
				}
				else if (typeSol == 7) {
					x3 = root5[BestRoot];
				}
				else if (typeSol == 8) {
					x3 = root6[BestRoot];
				}
				p = sqrt((k2 * x4) / (c1122 * x3));
				if (neg) {
					p = -p;
				}

				x1 = p * c1 * x3;
				x2 = p * c2 * x3;
			}
			else {
				if (typeSol == 2) {
					x3 = x3max;
					p = root2[BestRoot];
				}
				else if (typeSol == 3) {
					x3 = x3min;
					p = root3[BestRoot];
				}
				else if (typeSol == 4) {
					p = root4[BestRoot];
					x3 = -(c1122 * p + 2 * c3) / (2 * (c1122 * p * p + 2));
					x3 = (x3max - x3) * (x3 > x3max) + (x3min - x3) * (x3min > x3) + x3;
				}
				x1 = p * c1 * x3;
				x2 = p * c2 * x3;
				x4 = (x1 * x1 + x2 * x2) / (x3 * k2);
				x4 = (x4max - x4) * (x4 > x4max) + x4;
			}
		}

		// X =  {Pi, Qi, vi, li, vAi, (pn, qn), (Pci, Qci, lci) for all child Ci}


		if (typeSol) {
			if (x4 > x4max) {
				std::cout << "probleme bus " << i << " " << x4max << " " << goodSol << " " << typeSol << std::endl;
				std::cout << "x : " << x1 << " " << x2 << " " << x3 * k2 << " " << x4 << " " << std::endl;
			}

			X[i].set(0, 0, x1);
			X[i].set(1, 0, x2);
			X[i].set(2, 0, x3 * k2);
			X[i].set(3, 0, x4);
		}
		else {
			std::cout << "pas de solution, pas de changement" << std::endl;
			std::cout << "probleme bus " << i << " " << x4max << " " << goodSol << " " << typeSol << std::endl;
			std::cout << "x : " << x1 << " " << x2 << " " << x3 * k2 << " " << x4 << " " << std::endl;

		}
		
		
		//std::cout << "x F : " << x1 << " " << x2 << " " << x3*k2 << " " << x4 << " " << gamma << std::endl;
		
		
		// pn & qn
		int Nb = _nAgentByBus.get(i, 0);
		int begin = _CoresAgentBusBegin.get(i, 0);
		for (int In = 0; In < Nb; In++) {
			int n = _CoresAgentBus.get(In + begin, 0);
			float ub = Pmax.get(n, 0);
			float lb = Pmin.get(n, 0);
			float pn = (_rho * Chat[i].get(4 + 2 * In, 0) - Cost2.get(n, 0)) / (Cost1.get(n, 0) + _rho);
			pn = ub * (ub < pn) + lb * (lb > pn) + pn * (pn >= lb) * (pn <= ub);

			ub = Pmax.get(n + _nAgent, 0);
			lb = Pmin.get(n + _nAgent, 0);
			float qn = (_rho * Chat[i].get(5 + 2 * In, 0) - Cost2.get(n + _nAgent, 0)) / (Cost1.get(n + _nAgent, 0) + _rho);
			qn = ub * (ub < qn) + lb * (lb > qn) + qn * (qn >= lb) * (qn <= ub);
			
			// pn & qn
			X[i].set(5 + 2 * In, 0, pn);
			X[i].set(6 + 2 * In, 0, qn);
		}

	}

	//bus fictif
	float pn = (_rho * Chat[_nBus].get(0, 0) - Cost2.get(0, 0)) / (Cost1.get(0, 0) + _rho);
	float qn = (_rho * Chat[_nBus].get(1, 0) - Cost2.get(_nAgent, 0)) / (Cost1.get(_nAgent, 0) + _rho);


	switch (losstype)
	{
	case LossType::CURRENT:
		X[_nBus].set(0, 0, pn);
		X[_nBus].set(1, 0, qn);
		break;
	case LossType::POWER:
		X[_nBus].set(0, 0, pn);
		X[_nBus].set(_nAgent, 0, qn);
		break;
	}
*/
	
}

void OPFADMMConsGPU::updateXWOCurrent()
{
	updateXOPFADMMCons << <_nBus, _blockSizeSmall >> > (X._matrixGPU, Pn._matrixGPU, Chat._matrixGPU, VoltageLimit._matrixGPU, _nAgentByBus._matrixGPU, nChild._matrixGPU, _indiceBusBegin._matrixGPU, _CoresChatBegin._matrixGPU,
		_CoresAgentBusBegin._matrixGPU, _CoresAgentBus._matrixGPU, Cost1._matrixGPU, Cost2._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, _rho, losstype, _nBus, _nAgent, Lagrange);

}

void OPFADMMConsGPU::updateXWOCurrentOnCPU()
{
	X.transferCPU();
	Chat.transferCPU();
	double x1, x2, x3, x4, c1, c2, c3, c4, lambdaLo, lambdaUp, x3min, x3max, gamma, k2; // double peut �tre necessaire
	double c1122; // c3 : voltage -> indice + 3, c4 : current -> indice + 2;
	
	


	for (int bus = 1; bus < _nBus; bus++) {

		int typeSol = 0;
		int BestRoot = 0;
		double bestGamma = -1;
		double p = 0;

		int nRoot = 0;

		int begining = indiceBusBeginCPU.get(bus, 0);
		int nC = nChildCPU.get(bus,0);
		int beginChat = CoresChatBeginCPU.get(bus, 0);
		bool goodSol = false;
		k2 = sqrt(2.0 / (nC + 1));

		c1 = -2 * Chat.get(beginChat, 0);
		c2 = -2 * Chat.get(beginChat + 1, 0);
		c4 = -2 * Chat.get(beginChat + 2, 0);
		c3 = -2 * Chat.get(beginChat + 3, 0) / k2;

		c1122 = c1 * c1 + c2 * c2;
		x3min = VoltageLimitCPU.get(0, bus);
		x3max = VoltageLimitCPU.get(1, bus);

		// case without constraint

		x1 = -c1 / 2;
		x2 = -c2 / 2;
		x3 = -c3 / 2;
		x4 = -c4 / 2;

		lambdaUp = 0;
		lambdaLo = 0;
		if (x3 < x3min) {
			x3 = x3min;
			lambdaLo = (2 * x3 + c3);
		}
		else if (x3 > x3max) {
			x3 = x3max;
			lambdaUp = -(2 * x3 + c3);
		}
		gamma = k2 * x4 - (x1 * x1 + x2 * x2) / x3; // ce n'est pas vraiment gamma, doit être positif

		if (gamma >= 0) {
			// the solution is good !
			goodSol = true;
		}
		else {
			if (c1122 == 0) {
				x4 = 0;
				goodSol = true;
			}
			if (gamma > bestGamma) {
				typeSol = 1;
				bestGamma = gamma;
			}
		}
		//}
		if (!goodSol) {
			x3 = x3max;

			coefPoly2[0] = 2 * (c4 / (k2 * x3) + 1);
			coefPoly2[1] = 1 / x3;
			coefPoly2[0] = coefPoly2[0] * k2 * k2 / (4 * c1122);
			coefPoly2[1] = coefPoly2[1] * k2 * k2 / (4 * c1122);
			nRoot = resolveRealPolynome3without2term(root2, coefPoly2);

			for (int n = 0; n < nRoot; n++) {
				p = root2[n];

				x1 = p * c1 * x3;
				x2 = p * c2 * x3;
				x4 = (x1 * x1 + x2 * x2) / (x3 * k2);
				gamma = (2 * x4 + c4) / k2;
				lambdaUp = -(2 * x3 + c3 - gamma * (x1 * x1 + x2 * x2) / (x3 * x3));

				if (gamma >= 0 && lambdaUp >= 0) {
					goodSol = true;
					break;
				}
				if (gamma > bestGamma && lambdaUp > bestGamma) {
					typeSol = 2;
					bestGamma = MYMIN(gamma, lambdaUp);
					BestRoot = n;
				}

			}
		}
		// case x3 = x3min lambdaUp = 0
		if (!goodSol) {
			x3 = x3min;

			coefPoly2[0] = 2 * (c4 / (k2 * x3) + 1);
			coefPoly2[1] = 1 / x3;
			coefPoly2[0] = coefPoly2[0] * k2 * k2 / (4 * c1122);
			coefPoly2[1] = coefPoly2[1] * k2 * k2 / (4 * c1122);
			nRoot = resolveRealPolynome3without2term(root3, coefPoly2);

			for (int n = 0; n < nRoot; n++) {
				p = root3[n];
				x1 = p * c1 * x3;
				x2 = p * c2 * x3;
				x4 = (x1 * x1 + x2 * x2) / (x3 * k2);
				gamma = (2 * x4 + c4) / k2;
				lambdaLo = (2 * x3 + c3 - gamma * (x1 * x1 + x2 * x2) / (x3 * x3));

				if (gamma >= 0 && lambdaLo >= 0) {
					// the solution is good !
					goodSol = true;
					break;
				}
				if (gamma > bestGamma && lambdaLo > bestGamma) {
					typeSol = 3;
					bestGamma = MYMIN(gamma, lambdaLo);
					BestRoot = n;
				}
			}
		}
		// case xmin<x3<xmax lambdaLo = 0 lambdaUp = 0
		if (!goodSol) {

			coefPoly3[0] = c1122 / k2 * (2 * c3 / k2 - c4);
			coefPoly3[1] = (c3 - 2 * c4 / k2);
			coefPoly3[2] = -1;
			coefPoly3[0] = coefPoly3[0] * k2 * k2 / (c1122 * c1122);
			coefPoly3[1] = coefPoly3[1] * k2 * k2 / (c1122 * c1122);
			coefPoly3[2] = coefPoly3[2] * k2 * k2 / (c1122 * c1122);

			nRoot = resvolveRealPolynome4without2term(root4, coefPoly3);

			for (int n = 0; n < nRoot; n++) {
				p = root4[n];
				x3 = -(c1122 * p + 2 * c3) / (2 * (c1122 * p * p + 2));
				x1 = p * c1 * x3;
				x2 = p * c2 * x3;
				x4 = (x1 * x1 + x2 * x2) / (x3 * k2);
				gamma = (2 * x4 + c4) / k2;

				if (gamma >= 0 && x3 <= x3max && x3 >= x3min) {
					// the solution is good !
					goodSol = true;
					break;
				}
				if (gamma > bestGamma && (x3max - x3) > bestGamma && (x3 - x3min) > bestGamma) {
					typeSol = 4;
					bestGamma = MYMIN(MYMIN(gamma, (x3max - x3)), (x3 - x3min));
					BestRoot = n;
				}
			}
		}
		if (!goodSol) {

			if (typeSol == 1) {
				// case without constraint
				x1 = -c1 / 2;
				x2 = -c2 / 2;
				x3 = -c3 / 2;
				x3 = (x3max - x3) * (x3 > x3max) + (x3min - x3) * (x3min > x3) + x3;
				x4 = -c4 / 2; // ou  (x1 * x1 + x2 * x2) / (k2 * x3)
			}
			else {
				if (typeSol == 2) {
					x3 = x3max;
					p = root2[BestRoot];
				}
				else if (typeSol == 3) {
					x3 = x3min;
					p = root3[BestRoot];
				}
				else if (typeSol == 4) {
					p = root4[BestRoot];
					x3 = -(c1122 * p + 2 * c3) / (2 * (c1122 * p * p + 2));
					x3 = (x3max - x3) * (x3 > x3max) + (x3min - x3) * (x3min > x3) + x3;
				}
				x1 = p * c1 * x3;
				x2 = p * c2 * x3;
				x4 = (x1 * x1 + x2 * x2) / (x3 * k2);
			}
		}

		X.set(begining, 0, x1);
		X.set(begining + 1, 0, x2);
		X.set(begining + 2, 0, x4);
		X.set(begining + 3, 0, x3 *k2);
	}



	X.transferGPU();
	Chat.transferGPU();


	updateXPnOPFADMMCons << <_nBusWLoss, _blockSizeSmall >> > (X._matrixGPU, Pn._matrixGPU, Chat._matrixGPU, _nAgentByBus._matrixGPU, _indiceBusBegin._matrixGPU, _CoresChatBegin._matrixGPU,
		_CoresAgentBusBegin._matrixGPU, _CoresAgentBus._matrixGPU, Cost1._matrixGPU, Cost2._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, _rho, losstype, _nBus, _nAgent);


}


void OPFADMMConsGPU::updateXWOCurrentOnCPUBis()
{
	X.transferCPU();
	Chat.transferCPU();
	double x1, x2, x3, x4, c1, c2, c3, c4, lambdaLo, lambdaUp, x3min, x3max, gamma, k2; // double peut �tre necessaire
	double c1122; // c3 : voltage -> indice + 3, c4 : current -> indice + 2;

	// résolution du polynome 4 sur GPU
	int nPoly = _nBus - 1;
	MatrixGPUD coefsGPU(4, nPoly);
	MatrixGPUD rootsGPU(4, nPoly, 0, 1);
	MatrixGPUD nRootGPU(nPoly, 1, 0, 1);

	for (int bus = 1; bus < _nBus; bus++) {
		int nC = nChildCPU.get(bus, 0);
		int beginChat = CoresChatBeginCPU.get(bus, 0);
		k2 = sqrt(2.0 / (nC + 1));
		c1 = -2 * Chat.get(beginChat, 0);
		c2 = -2 * Chat.get(beginChat + 1, 0);
		c4 = -2 * Chat.get(beginChat + 2, 0);
		c3 = -2 * Chat.get(beginChat + 3, 0) / k2;
		c1122 = c1 * c1 + c2 * c2;

		double aInv = k2 * k2 / (c1122 * c1122);
		double b = c1122 / k2 * (2 * c3 / k2 - c4);
		double d = (c3 - 2 * c4 / k2);
		double e = -1;

		b *= aInv;
		d *= aInv;
		e *= aInv;

		coefsGPU.set(0, bus - 1, b);
		coefsGPU.set(1, bus - 1, 0);
		coefsGPU.set(2, bus - 1, d);
		coefsGPU.set(3, bus - 1, e);
	}
	coefsGPU.transferGPU();
	resolveSeveralRealPolynome4WO2termGPULagrange << <_numBlocksB, _blockSizeSmall >> > (nRootGPU._matrixGPU, rootsGPU._matrixGPU, coefsGPU._matrixGPU, nPoly);
	nRootGPU.transferCPU();
	rootsGPU.transferCPU();


	for (int bus = 1; bus < _nBus; bus++) {

		int typeSol = 0;
		int BestRoot = 0;
		double bestGamma = -1;
		double p = 0;

		int nRoot = 0;

		int begining = indiceBusBeginCPU.get(bus, 0);
		int nC = nChildCPU.get(bus, 0);
		int beginChat = CoresChatBeginCPU.get(bus, 0);
		bool goodSol = false;
		k2 = sqrt(2.0 / (nC + 1));

		c1 = -2 * Chat.get(beginChat, 0);
		c2 = -2 * Chat.get(beginChat + 1, 0);
		c4 = -2 * Chat.get(beginChat + 2, 0);
		c3 = -2 * Chat.get(beginChat + 3, 0) / k2;

		c1122 = c1 * c1 + c2 * c2;
		x3min = VoltageLimitCPU.get(0, bus);
		x3max = VoltageLimitCPU.get(1, bus);

		// case without constraint

		x1 = -c1 / 2;
		x2 = -c2 / 2;
		x3 = -c3 / 2;
		x4 = -c4 / 2;

		lambdaUp = 0;
		lambdaLo = 0;
		if (x3 < x3min) {
			x3 = x3min;
			lambdaLo = (2 * x3 + c3);
		}
		else if (x3 > x3max) {
			x3 = x3max;
			lambdaUp = -(2 * x3 + c3);
		}
		gamma = k2 * x4 - (x1 * x1 + x2 * x2) / x3; // ce n'est pas vraiment gamma, doit être positif

		if (gamma >= 0) {
			// the solution is good !
			goodSol = true;
		}
		else {
			if (c1122 == 0) {
				x4 = 0;
				goodSol = true;
			}
			if (gamma > bestGamma) {
				typeSol = 1;
				bestGamma = gamma;
			}
		}
		//}
		if (!goodSol) {
			x3 = x3max;

			coefPoly2[0] = 2 * (c4 / (k2 * x3) + 1);
			coefPoly2[1] = 1 / x3;
			coefPoly2[0] = coefPoly2[0] * k2 * k2 / (4 * c1122);
			coefPoly2[1] = coefPoly2[1] * k2 * k2 / (4 * c1122);
			nRoot = resolveRealPolynome3without2term(root2, coefPoly2);

			for (int n = 0; n < nRoot; n++) {
				p = root2[n];

				x1 = p * c1 * x3;
				x2 = p * c2 * x3;
				x4 = (x1 * x1 + x2 * x2) / (x3 * k2);
				gamma = (2 * x4 + c4) / k2;
				lambdaUp = -(2 * x3 + c3 - gamma * (x1 * x1 + x2 * x2) / (x3 * x3));

				if (gamma >= 0 && lambdaUp >= 0) {
					goodSol = true;
					break;
				}
				if (gamma > bestGamma && lambdaUp > bestGamma) {
					typeSol = 2;
					bestGamma = MYMIN(gamma, lambdaUp);
					BestRoot = n;
				}

			}
		}
		// case x3 = x3min lambdaUp = 0
		if (!goodSol) {
			x3 = x3min;

			coefPoly2[0] = 2 * (c4 / (k2 * x3) + 1);
			coefPoly2[1] = 1 / x3;
			coefPoly2[0] = coefPoly2[0] * k2 * k2 / (4 * c1122);
			coefPoly2[1] = coefPoly2[1] * k2 * k2 / (4 * c1122);
			nRoot = resolveRealPolynome3without2term(root3, coefPoly2);

			for (int n = 0; n < nRoot; n++) {
				p = root3[n];
				x1 = p * c1 * x3;
				x2 = p * c2 * x3;
				x4 = (x1 * x1 + x2 * x2) / (x3 * k2);
				gamma = (2 * x4 + c4) / k2;
				lambdaLo = (2 * x3 + c3 - gamma * (x1 * x1 + x2 * x2) / (x3 * x3));

				if (gamma >= 0 && lambdaLo >= 0) {
					// the solution is good !
					goodSol = true;
					break;
				}
				if (gamma > bestGamma && lambdaLo > bestGamma) {
					typeSol = 3;
					bestGamma = MYMIN(gamma, lambdaLo);
					BestRoot = n;
				}
			}
		}
		// case xmin<x3<xmax lambdaLo = 0 lambdaUp = 0
		if (!goodSol) {

			/*coefPoly3[0] = c1122 / k2 * (2 * c3 / k2 - c4);
			coefPoly3[1] = (c3 - 2 * c4 / k2);
			coefPoly3[2] = -1;
			coefPoly3[0] = coefPoly3[0] * k2 * k2 / (c1122 * c1122);
			coefPoly3[1] = coefPoly3[1] * k2 * k2 / (c1122 * c1122);
			coefPoly3[2] = coefPoly3[2] * k2 * k2 / (c1122 * c1122);

			nRoot = resvolveRealPolynome4without2term(root4, coefPoly3);*/

			nRoot = nRootGPU.get(bus - 1, 0);

			for (int n = 0; n < nRoot; n++) {
				root4[n] = rootsGPU.get(n, bus - 1);
				p = root4[n];
				x3 = -(c1122 * p + 2 * c3) / (2 * (c1122 * p * p + 2));
				x1 = p * c1 * x3;
				x2 = p * c2 * x3;
				x4 = (x1 * x1 + x2 * x2) / (x3 * k2);
				gamma = (2 * x4 + c4) / k2;

				if (gamma >= 0 && x3 <= x3max && x3 >= x3min) {
					// the solution is good !
					goodSol = true;
					break;
				}
				if (gamma > bestGamma && (x3max - x3) > bestGamma && (x3 - x3min) > bestGamma) {
					typeSol = 4;
					bestGamma = MYMIN(MYMIN(gamma, (x3max - x3)), (x3 - x3min));
					BestRoot = n;
				}
			}
		}
		if (!goodSol) {

			if (typeSol == 1) {
				// case without constraint
				x1 = -c1 / 2;
				x2 = -c2 / 2;
				x3 = -c3 / 2;
				x3 = (x3max - x3) * (x3 > x3max) + (x3min - x3) * (x3min > x3) + x3;
				x4 = -c4 / 2; // ou  (x1 * x1 + x2 * x2) / (k2 * x3)
			}
			else {
				if (typeSol == 2) {
					x3 = x3max;
					p = root2[BestRoot];
				}
				else if (typeSol == 3) {
					x3 = x3min;
					p = root3[BestRoot];
				}
				else if (typeSol == 4) {
					p = root4[BestRoot];
					x3 = -(c1122 * p + 2 * c3) / (2 * (c1122 * p * p + 2));
					x3 = (x3max - x3) * (x3 > x3max) + (x3min - x3) * (x3min > x3) + x3;
				}
				x1 = p * c1 * x3;
				x2 = p * c2 * x3;
				x4 = (x1 * x1 + x2 * x2) / (x3 * k2);
			}
		}

		X.set(begining, 0, x1);
		X.set(begining + 1, 0, x2);
		X.set(begining + 2, 0, x4);
		X.set(begining + 3, 0, x3 * k2);
	}



	X.transferGPU();
	Chat.transferGPU();


	updateXPnOPFADMMCons << <_nBusWLoss, _blockSizeSmall >> > (X._matrixGPU, Pn._matrixGPU, Chat._matrixGPU, _nAgentByBus._matrixGPU, _indiceBusBegin._matrixGPU, _CoresChatBegin._matrixGPU,
		_CoresAgentBusBegin._matrixGPU, _CoresAgentBus._matrixGPU, Cost1._matrixGPU, Cost2._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, _rho, losstype, _nBus, _nAgent);


}

void OPFADMMConsGPU::updateXWOCurrentOnCPUBis(bool first)
{
	X.transferCPU();
	Chat.transferCPU();
	double x1, x2, x3, x4, c1, c2, c3, c4, lambdaLo, lambdaUp, x3min, x3max, gamma, k2; // double peut �tre necessaire
	double c1122; // c3 : voltage -> indice + 3, c4 : current -> indice + 2;

	// résolution du polynome 4 sur GPU
	int nPoly = _nBus - 1;
	MatrixGPUD coefsGPU(4, nPoly);
	MatrixGPUD rootsGPU(3, nPoly, 0, 1);
	MatrixGPUD nRootGPU(nPoly, 1, 0, 1);

	for (int bus = 1; bus < _nBus; bus++) {
		int nC = nChildCPU.get(bus, 0);
		int beginChat = CoresChatBeginCPU.get(bus, 0);
		k2 = sqrt(2.0 / (nC + 1));
		c1 = -2 * Chat.get(beginChat, 0);
		c2 = -2 * Chat.get(beginChat + 1, 0);
		c4 = -2 * Chat.get(beginChat + 2, 0);
		c3 = -2 * Chat.get(beginChat + 3, 0) / k2;
		c1122 = c1 * c1 + c2 * c2;
		x3min = VoltageLimitCPU.get(0, bus);
		x3max = VoltageLimitCPU.get(1, bus);

		if (first){
			x3 = x3max;
		}
		else {
			x3 = x3min;
		}
		
		double aInv = k2 * k2 / (4 * c1122);
		double p = 2 * (c4 / (k2 * x3) + 1);
		double q = 1 / x3;

		

		p *= aInv;
		q *= aInv;
	
		coefsGPU.set(0, bus - 1, 1);
		coefsGPU.set(1, bus - 1, 0);
		coefsGPU.set(2, bus - 1, p);
		coefsGPU.set(3, bus - 1, q);
	}
	coefsGPU.transferGPU();
	resolveSeveralRealPolynome3termGPU << <_numBlocksB, _blockSizeSmall >> > (nRootGPU._matrixGPU, rootsGPU._matrixGPU, coefsGPU._matrixGPU, nPoly);
	nRootGPU.transferCPU();
	rootsGPU.transferCPU();


	for (int bus = 1; bus < _nBus; bus++) {

		int typeSol = 0;
		int BestRoot = 0;
		double bestGamma = -1;
		double p = 0;

		int nRoot = 0;

		int begining = indiceBusBeginCPU.get(bus, 0);
		int nC = nChildCPU.get(bus, 0);
		int beginChat = CoresChatBeginCPU.get(bus, 0);
		bool goodSol = false;
		k2 = sqrt(2.0 / (nC + 1));

		c1 = -2 * Chat.get(beginChat, 0);
		c2 = -2 * Chat.get(beginChat + 1, 0);
		c4 = -2 * Chat.get(beginChat + 2, 0);
		c3 = -2 * Chat.get(beginChat + 3, 0) / k2;

		c1122 = c1 * c1 + c2 * c2;
		x3min = VoltageLimitCPU.get(0, bus);
		x3max = VoltageLimitCPU.get(1, bus);

		// case without constraint

		x1 = -c1 / 2;
		x2 = -c2 / 2;
		x3 = -c3 / 2;
		x4 = -c4 / 2;

		lambdaUp = 0;
		lambdaLo = 0;
		if (x3 < x3min) {
			x3 = x3min;
			lambdaLo = (2 * x3 + c3);
		}
		else if (x3 > x3max) {
			x3 = x3max;
			lambdaUp = -(2 * x3 + c3);
		}
		gamma = k2 * x4 - (x1 * x1 + x2 * x2) / x3; // ce n'est pas vraiment gamma, doit être positif

		if (gamma >= 0) {
			// the solution is good !
			goodSol = true;
		}
		else {
			if (c1122 == 0) {
				x4 = 0;
				goodSol = true;
			}
			if (gamma > bestGamma) {
				typeSol = 1;
				bestGamma = gamma;
			}
		}
		//}
		if (!goodSol) {
			x3 = x3max;

			if (first) {
				nRoot = nRootGPU.get(bus - 1, 0);
				for (int n = 0; n < nRoot; n++) {
					root2[n] = rootsGPU.get(n, bus - 1);
				}
			}
			else {
				coefPoly2[0] = 2 * (c4 / (k2 * x3) + 1);
				coefPoly2[1] = 1 / x3;
				coefPoly2[0] = coefPoly2[0] * k2 * k2 / (4 * c1122);
				coefPoly2[1] = coefPoly2[1] * k2 * k2 / (4 * c1122);
				nRoot = resolveRealPolynome3without2term(root2, coefPoly2);
			}
			

			for (int n = 0; n < nRoot; n++) {
				p = root2[n];

				x1 = p * c1 * x3;
				x2 = p * c2 * x3;
				x4 = (x1 * x1 + x2 * x2) / (x3 * k2);
				gamma = (2 * x4 + c4) / k2;
				lambdaUp = -(2 * x3 + c3 - gamma * (x1 * x1 + x2 * x2) / (x3 * x3));

				if (gamma >= 0 && lambdaUp >= 0) {
					goodSol = true;
					break;
				}
				if (gamma > bestGamma && lambdaUp > bestGamma) {
					typeSol = 2;
					bestGamma = MYMIN(gamma, lambdaUp);
					BestRoot = n;
				}

			}
		}
		// case x3 = x3min lambdaUp = 0
		if (!goodSol) {
			x3 = x3min;

			if (!first) {
				nRoot = nRootGPU.get(bus - 1, 0);
				for (int n = 0; n < nRoot; n++) {
					root3[n] = rootsGPU.get(n, bus - 1);
				}
			}
			else {
				coefPoly2[0] = 2 * (c4 / (k2 * x3) + 1);
				coefPoly2[1] = 1 / x3;
				coefPoly2[0] = coefPoly2[0] * k2 * k2 / (4 * c1122);
				coefPoly2[1] = coefPoly2[1] * k2 * k2 / (4 * c1122);
				nRoot = resolveRealPolynome3without2term(root2, coefPoly2);
			}

			for (int n = 0; n < nRoot; n++) {
				p = root3[n];
				x1 = p * c1 * x3;
				x2 = p * c2 * x3;
				x4 = (x1 * x1 + x2 * x2) / (x3 * k2);
				gamma = (2 * x4 + c4) / k2;
				lambdaLo = (2 * x3 + c3 - gamma * (x1 * x1 + x2 * x2) / (x3 * x3));

				if (gamma >= 0 && lambdaLo >= 0) {
					// the solution is good !
					goodSol = true;
					break;
				}
				if (gamma > bestGamma && lambdaLo > bestGamma) {
					typeSol = 3;
					bestGamma = MYMIN(gamma, lambdaLo);
					BestRoot = n;
				}
			}
		}
		// case xmin<x3<xmax lambdaLo = 0 lambdaUp = 0
		if (!goodSol) {

			coefPoly3[0] = c1122 / k2 * (2 * c3 / k2 - c4);
			coefPoly3[1] = (c3 - 2 * c4 / k2);
			coefPoly3[2] = -1;
			coefPoly3[0] = coefPoly3[0] * k2 * k2 / (c1122 * c1122);
			coefPoly3[1] = coefPoly3[1] * k2 * k2 / (c1122 * c1122);
			coefPoly3[2] = coefPoly3[2] * k2 * k2 / (c1122 * c1122);

			nRoot = resvolveRealPolynome4without2term(root4, coefPoly3, Lagrange);/**/

			

			for (int n = 0; n < nRoot; n++) {
				p = root4[n];
				x3 = -(c1122 * p + 2 * c3) / (2 * (c1122 * p * p + 2));
				x1 = p * c1 * x3;
				x2 = p * c2 * x3;
				x4 = (x1 * x1 + x2 * x2) / (x3 * k2);
				gamma = (2 * x4 + c4) / k2;

				if (gamma >= 0 && x3 <= x3max && x3 >= x3min) {
					// the solution is good !
					goodSol = true;
					break;
				}
				if (gamma > bestGamma && (x3max - x3) > bestGamma && (x3 - x3min) > bestGamma) {
					typeSol = 4;
					bestGamma = MYMIN(MYMIN(gamma, (x3max - x3)), (x3 - x3min));
					BestRoot = n;
				}
			}
		}
		if (!goodSol) {

			if (typeSol == 1) {
				// case without constraint
				x1 = -c1 / 2;
				x2 = -c2 / 2;
				x3 = -c3 / 2;
				x3 = (x3max - x3) * (x3 > x3max) + (x3min - x3) * (x3min > x3) + x3;
				x4 = -c4 / 2; // ou  (x1 * x1 + x2 * x2) / (k2 * x3)
			}
			else {
				if (typeSol == 2) {
					x3 = x3max;
					p = root2[BestRoot];
				}
				else if (typeSol == 3) {
					x3 = x3min;
					p = root3[BestRoot];
				}
				else if (typeSol == 4) {
					p = root4[BestRoot];
					x3 = -(c1122 * p + 2 * c3) / (2 * (c1122 * p * p + 2));
					x3 = (x3max - x3) * (x3 > x3max) + (x3min - x3) * (x3min > x3) + x3;
				}
				x1 = p * c1 * x3;
				x2 = p * c2 * x3;
				x4 = (x1 * x1 + x2 * x2) / (x3 * k2);
			}
		}

		X.set(begining, 0, x1);
		X.set(begining + 1, 0, x2);
		X.set(begining + 2, 0, x4);
		X.set(begining + 3, 0, x3 * k2);
	}



	X.transferGPU();
	Chat.transferGPU();


	updateXPnOPFADMMCons << <_nBusWLoss, _blockSizeSmall >> > (X._matrixGPU, Pn._matrixGPU, Chat._matrixGPU, _nAgentByBus._matrixGPU, _indiceBusBegin._matrixGPU, _CoresChatBegin._matrixGPU,
		_CoresAgentBusBegin._matrixGPU, _CoresAgentBus._matrixGPU, Cost1._matrixGPU, Cost2._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, _rho, losstype, _nBus, _nAgent);


}


void OPFADMMConsGPU::updateMu()
{
	updateMUGPU << <_numBlocksH, _blockSize >> > (Mu._matrixGPU, Y._matrixGPU, X._matrixGPU, _rho, _sizeOPFADMMConsTotal);

}



float OPFADMMConsGPU::getPLoss()
{
	_Ploss = Y.get(_debutloss, 0, false);

	return _Ploss;
}

float OPFADMMConsGPU::getQLoss()
{
	int indice = 1;
	if (losstype == LossType::POWER) {
		indice = _nAgent;
	}

	_Qloss = Y.get(_debutloss + indice, 0, false);

	return _Qloss;
}

void OPFADMMConsGPU::computeLoss()
{

	int numBlock = 1;
	switch (_blockSize) {
	case 512:
		ComputeLoss<512> << <numBlock, _blockSize >> > (X._matrixGPU, Pn._matrixGPU, _indiceBusBegin._matrixGPU, ZsRe._matrixGPU, ZsIm._matrixGPU, losstype, _nAgent, _nBus);
		break;
	case 256:
		ComputeLoss<256> << <numBlock, _blockSize >> > (X._matrixGPU, Pn._matrixGPU, _indiceBusBegin._matrixGPU, ZsRe._matrixGPU, ZsIm._matrixGPU, losstype, _nAgent, _nBus);
		break;
	case 128:
		ComputeLoss<128> << <numBlock, _blockSize >> > (X._matrixGPU, Pn._matrixGPU, _indiceBusBegin._matrixGPU, ZsRe._matrixGPU, ZsIm._matrixGPU, losstype, _nAgent, _nBus);
		break;
	case 64:
		ComputeLoss< 64> << <numBlock, _blockSize >> > (X._matrixGPU, Pn._matrixGPU, _indiceBusBegin._matrixGPU, ZsRe._matrixGPU, ZsIm._matrixGPU, losstype, _nAgent, _nBus);
		break;
	case 32:
		ComputeLoss< 32> << <numBlock, _blockSize >> > (X._matrixGPU, Pn._matrixGPU, _indiceBusBegin._matrixGPU, ZsRe._matrixGPU, ZsIm._matrixGPU, losstype, _nAgent, _nBus);
		break;
	case 16:
		ComputeLoss< 16> << <numBlock, _blockSize >> > (X._matrixGPU, Pn._matrixGPU, _indiceBusBegin._matrixGPU, ZsRe._matrixGPU, ZsIm._matrixGPU, losstype, _nAgent, _nBus);
		break;
	case  8:
		ComputeLoss<  8> << <numBlock, _blockSize >> > (X._matrixGPU, Pn._matrixGPU, _indiceBusBegin._matrixGPU, ZsRe._matrixGPU, ZsIm._matrixGPU, losstype, _nAgent, _nBus);
		break;
	case  4:
		ComputeLoss<  4> << <numBlock, _blockSize >> > (X._matrixGPU, Pn._matrixGPU, _indiceBusBegin._matrixGPU, ZsRe._matrixGPU, ZsIm._matrixGPU, losstype, _nAgent, _nBus);
		break;
	case  2:
		ComputeLoss<  2> << <numBlock, _blockSize >> > (X._matrixGPU, Pn._matrixGPU, _indiceBusBegin._matrixGPU, ZsRe._matrixGPU, ZsIm._matrixGPU, losstype, _nAgent, _nBus);
		break;
	case  1:
		ComputeLoss<  1> << <numBlock, _blockSize >> > (X._matrixGPU, Pn._matrixGPU, _indiceBusBegin._matrixGPU, ZsRe._matrixGPU, ZsIm._matrixGPU, losstype, _nAgent, _nBus);
		break;
	}
}

void OPFADMMConsGPU::updateChat()
{
	int numBlock = _nBusWLoss;
	switch (_blockSizeSmall) {
	case 512:
		updateChatGPU4<512> << <numBlock, _blockSizeSmall >> > (Chat._matrixGPU, Y._matrixGPU, Mu._matrixGPU, nChild._matrixGPU, Ancestor._matrixGPU, PosChild._matrixGPU, Childs._matrixGPU, _indiceBusBegin._matrixGPU, _indiceChildBegin._matrixGPU, _CoresChatBegin._matrixGPU, _CoresAgentBusBegin._matrixGPU, _CoresAgentBus._matrixGPU, _nAgentByBus._matrixGPU, _rho, losstype, _nBus);
		break;
	case 256:
		updateChatGPU4<256> << <numBlock, _blockSizeSmall >> > (Chat._matrixGPU, Y._matrixGPU, Mu._matrixGPU, nChild._matrixGPU, Ancestor._matrixGPU, PosChild._matrixGPU, Childs._matrixGPU, _indiceBusBegin._matrixGPU, _indiceChildBegin._matrixGPU, _CoresChatBegin._matrixGPU, _CoresAgentBusBegin._matrixGPU, _CoresAgentBus._matrixGPU, _nAgentByBus._matrixGPU, _rho, losstype, _nBus);
		break;
	case 128:
		updateChatGPU4<128> << <numBlock, _blockSizeSmall >> > (Chat._matrixGPU, Y._matrixGPU, Mu._matrixGPU, nChild._matrixGPU, Ancestor._matrixGPU, PosChild._matrixGPU, Childs._matrixGPU, _indiceBusBegin._matrixGPU, _indiceChildBegin._matrixGPU, _CoresChatBegin._matrixGPU, _CoresAgentBusBegin._matrixGPU, _CoresAgentBus._matrixGPU, _nAgentByBus._matrixGPU, _rho, losstype, _nBus);
		break;
	case 64:
		updateChatGPU4< 64> << <numBlock, _blockSizeSmall >> > (Chat._matrixGPU, Y._matrixGPU, Mu._matrixGPU, nChild._matrixGPU, Ancestor._matrixGPU, PosChild._matrixGPU, Childs._matrixGPU, _indiceBusBegin._matrixGPU, _indiceChildBegin._matrixGPU, _CoresChatBegin._matrixGPU, _CoresAgentBusBegin._matrixGPU, _CoresAgentBus._matrixGPU, _nAgentByBus._matrixGPU, _rho, losstype, _nBus);
		break;
	case 32:
		updateChatGPU4< 32> << <numBlock, _blockSizeSmall >> > (Chat._matrixGPU, Y._matrixGPU, Mu._matrixGPU, nChild._matrixGPU, Ancestor._matrixGPU, PosChild._matrixGPU, Childs._matrixGPU, _indiceBusBegin._matrixGPU, _indiceChildBegin._matrixGPU, _CoresChatBegin._matrixGPU, _CoresAgentBusBegin._matrixGPU, _CoresAgentBus._matrixGPU, _nAgentByBus._matrixGPU, _rho, losstype, _nBus);
		break;
	case 16:
		updateChatGPU4< 16> << <numBlock, _blockSizeSmall >> > (Chat._matrixGPU, Y._matrixGPU, Mu._matrixGPU, nChild._matrixGPU, Ancestor._matrixGPU, PosChild._matrixGPU, Childs._matrixGPU, _indiceBusBegin._matrixGPU, _indiceChildBegin._matrixGPU, _CoresChatBegin._matrixGPU, _CoresAgentBusBegin._matrixGPU, _CoresAgentBus._matrixGPU, _nAgentByBus._matrixGPU, _rho, losstype, _nBus);
		break;
	case  8:
		updateChatGPU4<  8> << <numBlock, _blockSizeSmall >> > (Chat._matrixGPU, Y._matrixGPU, Mu._matrixGPU, nChild._matrixGPU, Ancestor._matrixGPU, PosChild._matrixGPU, Childs._matrixGPU, _indiceBusBegin._matrixGPU, _indiceChildBegin._matrixGPU, _CoresChatBegin._matrixGPU, _CoresAgentBusBegin._matrixGPU, _CoresAgentBus._matrixGPU, _nAgentByBus._matrixGPU, _rho, losstype, _nBus);
		break;
	case  4:
		updateChatGPU4<  4> << <numBlock, _blockSizeSmall >> > (Chat._matrixGPU, Y._matrixGPU, Mu._matrixGPU, nChild._matrixGPU, Ancestor._matrixGPU, PosChild._matrixGPU, Childs._matrixGPU, _indiceBusBegin._matrixGPU, _indiceChildBegin._matrixGPU, _CoresChatBegin._matrixGPU, _CoresAgentBusBegin._matrixGPU, _CoresAgentBus._matrixGPU, _nAgentByBus._matrixGPU, _rho, losstype, _nBus);
		break;
	case  2:
		updateChatGPU4<  2> << <numBlock, _blockSizeSmall >> > (Chat._matrixGPU, Y._matrixGPU, Mu._matrixGPU, nChild._matrixGPU, Ancestor._matrixGPU, PosChild._matrixGPU, Childs._matrixGPU, _indiceBusBegin._matrixGPU, _indiceChildBegin._matrixGPU, _CoresChatBegin._matrixGPU, _CoresAgentBusBegin._matrixGPU, _CoresAgentBus._matrixGPU, _nAgentByBus._matrixGPU, _rho, losstype, _nBus);
		break;
	case  1:
		updateChatGPU4<  1> << <numBlock, _blockSizeSmall >> > (Chat._matrixGPU, Y._matrixGPU, Mu._matrixGPU, nChild._matrixGPU, Ancestor._matrixGPU, PosChild._matrixGPU, Childs._matrixGPU, _indiceBusBegin._matrixGPU, _indiceChildBegin._matrixGPU, _CoresChatBegin._matrixGPU, _CoresAgentBusBegin._matrixGPU, _CoresAgentBus._matrixGPU, _nAgentByBus._matrixGPU, _rho, losstype, _nBus);
		break;
	}

}

void OPFADMMConsGPU::CommunicationX()
{
/**/ // X = { Pi, Qi, vi, li, vAi, (pn, qn) (Pci, Qci, lci) for all child Ci }
	
	communicateX << <_nBusWLoss, _blockSize >> > (X._matrixGPU, nChild._matrixGPU, Ancestor._matrixGPU, Childs._matrixGPU, _indiceBusBegin._matrixGPU, _indiceChildBegin._matrixGPU, _nAgentByBus._matrixGPU, _CoresBusAgent._matrixGPU, PosAgent._matrixGPU, losstype, _nBus, _nAgent);

	
	


	
	//Y      (Pi, Qi, vi, li, pi, qi, vai, (pn, qn) Pji, Qji, lji)
	// Q udate in argmin 0.5yHy + Qy

	updateQ << <_numBlocksH, _blockSize >> > (Q._matrixGPU, X._matrixGPU, Mu._matrixGPU, _rho, _sizeOPFADMMConsTotal);


}


float OPFADMMConsGPU::updateRes(int indice) 
{
	float resS = Y.max2(&Ypre);
	float resR = Y.max2(&X);
	float resV = 0;
	
	float oldrho = _rho;
	resF.set(0, indice, resR);
	resF.set(1, indice, oldrho * resS);
	resF.set(2, indice, resV);

	/*std::cout << resS << " " << resR << std::endl;

	for (int i = 0; i < _nBus; i++) {
		std::cout << " Y " << std::endl;
		Y[i].display();
	}

	for (int i = 0; i < _nBus; i++) {
		std::cout << " X " << std::endl;
		X[i].display();
	}*/
	if (_tau > 1) {
		if (resR > _mu * resS) {
			
			_rho = _tau * _rho;
		 
			Hinv.divide(_tau);
			//std::cout << _iterGlobal << "rho augmente " << _rho << std::endl;
		}
		else if (resS > _mu * resR) {// rho = rho / tau_inc;
		
			_rho = _rho / _tau;
		
			Hinv.multiply(_tau);
			//std::cout << _tau << " " << _mu << std::endl;
			//std::cout << _iterGlobal << "rho diminue " << _rho << std::endl;
		}
	}
	


	return MYMAX(MYMAX(resV, oldrho * resS), resR);
}

float OPFADMMConsGPU::updateResRhoFixe(int indice)
{
	float resS = _rho * Y.max2(&Ypre);
	float resR = Y.max2(&X);
	float resV = 0;

	resF.set(0, indice, resR);
	resF.set(1, indice, resS);
	resF.set(2, indice, resV);

	return MYMAX(MYMAX(resV, resS), resR);
}

int OPFADMMConsGPU::feasiblePoint()
{
	bool mustTrans = false;
	if (X.getPos()) {
		X.transferCPU();
		_indiceBusBegin.transferCPU();
		mustTrans = true;
	}
	// X  (Pi, Qi, li, vi, pn..., qn..., vai, Pci ..., Qci... , lci...) !!!!!

	MatrixCPU test(_nBus, 1, -1);
	int counter = 0;
	for (int bus = 0; bus < _nBus; bus++) {
		int begin = _indiceBusBegin.get(bus, 0);
		float Si = X.get(begin, 0) * X.get(begin, 0) + X.get(begin + 1, 0) * X.get(begin + 1, 0);
		float li = X.get(begin + 2, 0);
		float vi = X.get(begin + 3, 0);
		float err = Si - li * vi;
		test.set(bus, 0, err);
		if (abs(err) > 0.0001) {
			counter++;
		}
	}
	//std::cout << " erreur sur la relaXation " << test.max2() << " " << counter << std::endl;
	//test.display();

	if (mustTrans) {
		X.transferGPU();
		_indiceBusBegin.transferGPU();
	}
	resF.set(2, (_iterGlobal - 1) / _stepG, test.max2());
	return counter;
}

void OPFADMMConsGPU::ComputePFromAgentToBus()
{
	int numBlock = _nBus;
	switch (_blockSizeSmall) {
	case 512:
		ComputePFromAgentToBusGPU<512> << <numBlock, _blockSizeSmall >> > (Pb._matrixGPU, Pbmin._matrixGPU, Pbmax._matrixGPU, CoresSoloBusAgent._matrixGPU, Pn._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, _CoresAgentBus._matrixGPU, _nAgentByBus._matrixGPU, _CoresAgentBusBegin._matrixGPU, _nAgent, _nBus);
		break;
	case 256:
		ComputePFromAgentToBusGPU<256> << <numBlock, _blockSizeSmall >> > (Pb._matrixGPU, Pbmin._matrixGPU, Pbmax._matrixGPU, CoresSoloBusAgent._matrixGPU, Pn._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, _CoresAgentBus._matrixGPU, _nAgentByBus._matrixGPU, _CoresAgentBusBegin._matrixGPU, _nAgent, _nBus);
		break;
	case 128:
		ComputePFromAgentToBusGPU<128> << <numBlock, _blockSizeSmall >> > (Pb._matrixGPU, Pbmin._matrixGPU, Pbmax._matrixGPU, CoresSoloBusAgent._matrixGPU, Pn._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, _CoresAgentBus._matrixGPU, _nAgentByBus._matrixGPU, _CoresAgentBusBegin._matrixGPU, _nAgent, _nBus);
		break;
	case 64:
		ComputePFromAgentToBusGPU< 64> << <numBlock, _blockSizeSmall >> > (Pb._matrixGPU, Pbmin._matrixGPU, Pbmax._matrixGPU, CoresSoloBusAgent._matrixGPU, Pn._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, _CoresAgentBus._matrixGPU, _nAgentByBus._matrixGPU, _CoresAgentBusBegin._matrixGPU, _nAgent, _nBus);
		break;
	case 32:
		ComputePFromAgentToBusGPU< 32> << <numBlock, _blockSizeSmall >> > (Pb._matrixGPU, Pbmin._matrixGPU, Pbmax._matrixGPU, CoresSoloBusAgent._matrixGPU, Pn._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, _CoresAgentBus._matrixGPU, _nAgentByBus._matrixGPU, _CoresAgentBusBegin._matrixGPU, _nAgent, _nBus);
		break;
	case 16:
		ComputePFromAgentToBusGPU< 16> << <numBlock, _blockSizeSmall >> > (Pb._matrixGPU, Pbmin._matrixGPU, Pbmax._matrixGPU, CoresSoloBusAgent._matrixGPU, Pn._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, _CoresAgentBus._matrixGPU, _nAgentByBus._matrixGPU, _CoresAgentBusBegin._matrixGPU, _nAgent, _nBus);
		break;
	case  8:
		ComputePFromAgentToBusGPU<  8> << <numBlock, _blockSizeSmall >> > (Pb._matrixGPU, Pbmin._matrixGPU, Pbmax._matrixGPU, CoresSoloBusAgent._matrixGPU, Pn._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, _CoresAgentBus._matrixGPU, _nAgentByBus._matrixGPU, _CoresAgentBusBegin._matrixGPU, _nAgent, _nBus);
		break;
	case  4:
		ComputePFromAgentToBusGPU<  4> << <numBlock, _blockSizeSmall >> > (Pb._matrixGPU, Pbmin._matrixGPU, Pbmax._matrixGPU, CoresSoloBusAgent._matrixGPU, Pn._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, _CoresAgentBus._matrixGPU, _nAgentByBus._matrixGPU, _CoresAgentBusBegin._matrixGPU, _nAgent, _nBus);
		break;
	case  2:
		ComputePFromAgentToBusGPU<  2> << <numBlock, _blockSizeSmall >> > (Pb._matrixGPU, Pbmin._matrixGPU, Pbmax._matrixGPU, CoresSoloBusAgent._matrixGPU, Pn._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, _CoresAgentBus._matrixGPU, _nAgentByBus._matrixGPU, _CoresAgentBusBegin._matrixGPU, _nAgent, _nBus);
		break;
	case  1:
		ComputePFromAgentToBusGPU<  1> << <numBlock, _blockSizeSmall >> > (Pb._matrixGPU, Pbmin._matrixGPU, Pbmax._matrixGPU, CoresSoloBusAgent._matrixGPU, Pn._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, _CoresAgentBus._matrixGPU, _nAgentByBus._matrixGPU, _CoresAgentBusBegin._matrixGPU, _nAgent, _nBus);
		break;
	}
}

MatrixCPU OPFADMMConsGPU::getPb(){
	MatrixCPU PbCPU;
	Pb.toMatCPU(PbCPU);
	return PbCPU;
}
MatrixCPU OPFADMMConsGPU::getPhi(){
	bool transferToDo = false;
	if(Y.getPos()){
		Y.transferCPU();
		_indiceBusBegin.transferCPU();
		transferToDo = true;
	}
	MatrixCPU Phi(2*_nLine, 1);
	
	for (int i = 0; i <_nLine; i++)
	{
		Phi.set(i,0, Y.get(_indiceBusBegin.get(i + 1,0) + 0, 0));
		Phi.set(i + _nLine,0, Y.get(_indiceBusBegin.get(i + 1,0) + 1, 0));
	}
	if(transferToDo){
		Y.transferGPU();
		_indiceBusBegin.transferGPU();
	}
	return Phi;
}
MatrixCPU OPFADMMConsGPU::getE(){
	bool transferToDo = false;
	if(Y.getPos()){
		Y.transferCPU();
		_indiceBusBegin.transferCPU();
		transferToDo = true;
	}
	MatrixCPU E(2*_nBus, 1);
	
	for (int i = 0; i <_nBus; i++)
	{
		E.set(i,0, Y.get(_indiceBusBegin.get(i, 0) + 2, 0));
		E.set(i + _nLine,0, Y.get(_indiceBusBegin.get(i, 0) + 3, 0));
	}
	if(transferToDo){
		Y.transferGPU();
		_indiceBusBegin.transferGPU();
	}
	return E;
}


void OPFADMMConsGPU::display() {

	
	X.transferCPU();
	Y.transferCPU();
	Mu.transferCPU();
	Pn.transferCPU();
	ZsRe.transferCPU();
	_indiceBusBegin.transferCPU();
	VoltageLimitReal.transferCPU();
	Pmin.transferCPU();
	Pmax.transferCPU();
	Pbmax.transferCPU();
	Pbmin.transferCPU();
	Pb.transferCPU();
	_CoresAgentBusBegin.transferCPU();
	_CoresAgentBus.transferCPU();
	_CoresBusAgent.transferCPU();
	Cost1.transferCPU();
	Cost2.transferCPU();
	


	if (_iterGlobal == 0) {
		std::cout << "algorithm not launch" << std::endl;
	}
	else if (_iterGlobal < _iterG) {
		std::cout << "method " << _name << " converged in " << _iterGlobal << " iterations." << std::endl;
		std::cout << "Converged in " << (float)timeOPF / CLOCKS_PER_SEC << " seconds" << std::endl;

	}
	else {
		std::cout << "method " << _name << " not converged in " << _iterGlobal << " iterations." << std::endl;
		std::cout << "time taken " << (float)timeOPF / CLOCKS_PER_SEC << " seconds" << std::endl;
	}
	std::cout << "The power error of this state is (constraint) " << resF.get(0, _iterGlobal / _stepG) << " and convergence " << resF.get(1, _iterGlobal / _stepG) << std::endl;
	std::cout << "===============================================================|" << std::endl;
	std::cout << "      System Summary                                           |" << std::endl;
	std::cout << "===============================================================|" << std::endl;
	std::cout << "Buses            " << _nBus << std::endl;
	std::cout << "Branches         " << _nLine << std::endl;
	std::cout << "Agent            " << _nAgent << std::endl;
	std::cout << "Ploss            " << getPLoss() << std::endl;
	std::cout << "Qloss            " << getQLoss() << std::endl;


	std::cout << std::endl << std::endl;

	std::cout << "========================================================================================================|" << std::endl;
	std::cout << "      Bus Data                                                                                          |" << std::endl;
	std::cout << "========================================================================================================|" << std::endl;
	std::cout << " Bus |    Voltage  |   Power = Generation  + Load    |                Mu voltage and power              |" << std::endl;
	std::cout << "  #  |     Mag(pu) |    P (pu)      |      Q (pu)    |     V (pu)     |      P (pu)    |      Q (pu)    |" << std::endl;
	std::cout << "-----|-------------|----------------|----------------|----------------|----------------|----------------|" << std::endl;


	for (int b = 0; b < _nBus; b++) {
		int begining = _indiceBusBegin.get(b, 0);
		std::cout << std::setw(5) << b << "|" << std::setw(12) << sqrt(X.get(begining + 3, 0)) << " |" << std::setw(16)
			<< Pb.get(b, 0) << "|" << std::setw(16) << Pb.get(b, 0)
			<< "|" << std::setw(16) << Mu.get(begining + 3, 0) << "|" << std::setw(16)
			<< Mu.get(begining, 0) << "|" << std::setw(16) << Mu.get(begining + 1, 0) << "|" << std::endl;

	}
	std::cout << std::endl << std::endl;
	std::cout << "========================================================================================================|" << std::endl;
	std::cout << "      Line Data                                                                                         |" << std::endl;
	std::cout << "========================================================================================================|" << std::endl;
	std::cout << " Line |    From     |    To      |                           Upstream flow                              |" << std::endl;
	std::cout << "  #   |    Bus      |    Bus     |    P (pu)      |    Q (pu)      |     l (pu)     |     Loss (pu)     |" << std::endl;
	std::cout << "------|-------------|------------|----------------|----------------|----------------|-------------------|" << std::endl;

	for (int l = 0; l < _nLine; l++) {
		int b = l + 1;
		int begining = _indiceBusBegin.get(b, 0);
		std::cout << std::setw(6) << l << "|" << std::setw(12) << CoresLineBusCPU.get(l, 0) << " |" << std::setw(12)
			<< CoresLineBusCPU.get(l, 1) << "|" << std::setw(16) << X.get(begining + 0, 0)
			<< "|" << std::setw(16) << X.get(begining + 1, 0) << "|" << std::setw(16)
			<< X.get(begining + 2, 0) << "|" << std::setw(19) << X.get(begining + 2, 0) * ZsRe.get(l, 0) << "|" << std::endl;
	}
	std::cout << std::endl << std::endl;
	std::cout << "========================================================================================================|" << std::endl;
	std::cout << "     Constraints                                                                                        |" << std::endl;
	std::cout << "========================================================================================================|" << std::endl;
	std::cout << " Bus | Voltage | Voltage | Voltage |        Power Injection          |          Power Injection         |" << std::endl;
	std::cout << "  #  | Mag(pu) | MIN(pu) |  MYMAX(pu)|  P (pu) | Pmin (pu) | Pmax (pu) |  Q (pu)  | Qmin (pu) | Qmax (pu) |" << std::endl;
	std::cout << "-----|---------|---------|---------|---------|-----------|-----------|----------|-----------|-----------|" << std::endl;


	for (int b = 0; b < _nBus; b++) {
		int begining = _indiceBusBegin.get(b, 0);
		std::cout << std::setw(5) << b << "|" << std::setw(8) << sqrt(Y.get(begining + 3, 0)) << " |" << std::setw(9)
			<< VoltageLimitReal.get(0, b) << "|" << std::setw(9) << VoltageLimitReal.get(1, b)
			<< "|" << std::setw(9) << Pb.get(b, 0) << "|" << std::setw(11)
			<< Pbmin.get(b, 0) << "|" << std::setw(11) << Pbmax.get(b, 0) << "|" << std::setw(10) << Pb.get(b + _nBus, 0)
			<< "|" << std::setw(11) << Pbmin.get(b + _nBus, 0) << "|" << std::setw(11) << Pbmax.get(b + _nBus, 0) << "|" << std::endl;

	}
	std::cout << std::endl << std::endl;
	std::cout << "========================================================================================================|" << std::endl;
	std::cout << "      Agent Data                                                                                        |" << std::endl;
	std::cout << "========================================================================================================|" << std::endl;
	std::cout << " Agent |  Bus  |  Cost   |  Cost   |        Power Injection          |          Power Injection         |" << std::endl;
	std::cout << "  #    |   #   |  a (pu) |  b (pu) |  P (pu) | Pmin (pu) | Pmax (pu) |  Q (pu)  | Qmin (pu) | Qmax (pu) |" << std::endl;
	std::cout << "-------|-------|---------|---------|---------|-----------|-----------|----------|-----------|-----------|" << std::endl;

	for (int n = 0; n < _nAgent; n++) {
		int b = _CoresBusAgent.get(n, 0);
		std::cout << std::setw(7) << n << "|" << std::setw(7) << b << "|" << std::setw(8) << Cost1.get(n, 0) << " |" << std::setw(9)
			<< Cost2.get(n, 0) << "|" << std::setw(9) << Pn.get(n, 0) << "|" << std::setw(11)
			<< Pmin.get(n, 0) << "|" << std::setw(11) << Pmax.get(n, 0) << "|" << std::setw(10) << Pn.get(n + _nAgent, 0)
			<< "|" << std::setw(11) << Pmin.get(n + _nAgent, 0) << "|" << std::setw(11) << Pmax.get(n + _nAgent, 0) << "|" << std::endl;
	}


	std::cout << "========================================================================================================|" << std::endl;
	std::cout << "                      END PRINT                                                                         |" << std::endl;
	std::cout << "========================================================================================================|" << std::endl;

}

template <unsigned int _blockSizeSmall>
__global__ void updateChatGPU4(float* Chat, float* Y, float* MU, float* nChild, float* Ancestor, float* posChild, float* Childs, float* indiceBusBegin, float* indiceChildBegin, float* CoresChatBegin, float* indiceAgentBegin, float* CoresAgentBus,  float* nAgentByBus, float _rho, int losstype, int nBus) {

	int bus = blockIdx.x;
	int index = threadIdx.x;
	int step = blockDim.x;


	int beginChat = CoresChatBegin[bus];

	__shared__ float shArr[_blockSizeSmall]; // c'est grand pour pas grand chose...

	int beginBus = indiceBusBegin[bus];
	int beginChild = (bus < (nBus - 1)) ? indiceChildBegin[bus] : 0;
	int childCount = nChild[bus];
	int AncestorIndex = Ancestor[bus];
	int nAgent = nAgentByBus[bus];
	int c = posChild[bus];
	int beginAgent = indiceAgentBegin[bus];
	int beginLoss = indiceBusBegin[nBus];
	
	float var = 0;
	int borne = 4 + 2 * nAgent;
	int divideVar = 2 + ((losstype == 1) && (index == 2)) * 1 - ((losstype == 1) && (index > 3)) * 1;

	if (bus < nBus) {
		if (index < borne) {
			//float Phat, Qhat, lhat, vihat, pnhat..., qnhat...;
			var = Y[beginBus + index] / (divideVar) - MU[beginBus + index] / (divideVar * _rho);
			
			if (index < 3 && bus > 0) {
				int childCountAi = nChild[AncestorIndex];
				int nAgentAi = nAgentByBus[AncestorIndex];
				int indiceAncBus = indiceBusBegin[AncestorIndex] + 4 + 2 * nAgentAi + 1 + childCountAi * index + c;
				//var = indiceAncBus;
				var += Y[indiceAncBus] / divideVar - MU[indiceAncBus] / (divideVar * _rho);
				var += ((index == 2) && (losstype == 1)) ? (Y[beginLoss + 2 + bus] / divideVar - MU[beginLoss + 2 + bus] / (divideVar * _rho)) : 0;

			}
			if ((index > 3) && (losstype==0)) {
				int offset = index >= 4 + nAgent ? 4 + nAgent : 4;
				int n = CoresAgentBus[ beginAgent + index - offset];
				var += Y[beginLoss + n] / (divideVar) - MU[beginLoss + n] / (divideVar * _rho);
			}
		}
		
		float vhat = 0;
		float muhat = 0;
		for (int i = index; i < childCount; i += step) {
			int Bus2 = Childs[beginChild + i];
			int indiceBusChild = indiceBusBegin[Bus2];
			int nAgent2 = nAgentByBus[Bus2];
			muhat += MU[indiceBusChild + 4 + 2 * nAgent2]; // pas du tout coalescent
			vhat += Y[indiceBusChild + 4 + 2 * nAgent2]; // pas du tout coalescent
		}
		shArr[index] = vhat / (childCount + 1) - muhat / (_rho * (childCount + 1));
		__syncthreads();
		for (int size = _blockSizeSmall / 2; size > 0; size /= 2) { //uniform
			if (index < size) {
				shArr[index] += shArr[index + size];
			}
			__syncthreads();
		}

		if (index < borne) {
			if (index == 3) {
				var = shArr[0] + Y[beginBus + 3] / (childCount + 1) - MU[beginBus + 3] / (_rho * (childCount + 1)); //shArr[0];
			}
			Chat[beginChat + index] = var; // coalescent  !!!!
		}
	}
	else { //bus des pertes
		if (index == 0) {
			float phat;
			float qhat;
			if (losstype == 0) {
				phat = Y[beginBus] - MU[beginBus] / _rho;
				qhat = Y[beginBus + nAgent] - MU[beginBus + nAgent] / _rho;
			}
			else {
				phat = Y[beginBus] - MU[beginBus] / _rho;
				qhat = Y[beginBus + 1] - MU[beginBus + 1] / _rho;
			}
			Chat[beginChat] = phat;
			Chat[beginChat + 1] = qhat;
		}
	}

	
}




/*
	for (int n = 1; n < _nAgent; n++) {
		float eta = etaSO.get(n, 0) +  0.5 * (Pn.get(n, 0) - Pmarket->get(n, 0));
		etaSO.set(n, 0, eta);
		eta = etaSO.get(n + _nAgent, 0) + 0.5 * (Pn.get(n + _nAgent, 0) - Pmarket->get(n + _nAgent, 0));
		etaSO.set(n + _nAgent, 0,  eta);
	}

	Cost2.add(&Pn, Pmarket);
	Cost2.set(0, 0, 0);
	Cost2.set(_nAgent, 0, 0);
	//Cost2.set(0, 0, _Ploss + Pmarket->get(0, 0));
	//Cost2.set(_nAgent, 0, _Qloss + Pmarket->get(_nAgent, 0));
	Cost2.multiply(-0.5);
	Cost2.add(&etaSO);
	Cost2.multiply(_rhoSO);

*/

__global__ void updateConsensusGPU(float* Cost2, float* etaSO, float* Pn, float* Pmarket, float _rhoSO, int nAgent) {

	int index = threadIdx.x + blockIdx.x * blockDim.x;
	int step = blockDim.x * gridDim.x;

	for (int agent = index; agent < nAgent; agent += step) {
		float eta = etaSO[agent] + 0.5 * (Pn[agent] - Pmarket[agent]);
		etaSO[agent] = eta;
		Cost2[agent] = _rhoSO * (eta - 0.5 * (Pn[agent] + Pmarket[agent]));
		eta = etaSO[agent + nAgent] + 0.5 * (Pn[agent + nAgent] - Pmarket[agent + nAgent]);
		etaSO[agent + nAgent] = eta;
		Cost2[agent + nAgent] = _rhoSO * (eta - 0.5 * (Pn[agent + nAgent] + Pmarket[agent + nAgent]));
	}


}