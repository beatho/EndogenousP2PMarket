#include "hip/hip_runtime.h"
#include "../head/TestADMMGPUConst1.cuh"
#define NSTEPLOCAL 5
#define NMAXPEERPERTRHREAD 5

int testADMMGPUConst1()
{
	int n = 1;

	if (!testADMMGPUConst1Contruct1()) return n;
	n++;
	
	if (!testADMMGPUConst1Contruct2()) return n;
	n++;
	
	if (!testADMMGPUConst1Contruct3()) return n;
	n++;
	
	if (!testADMMGPUConst1LAMBDA()) return n;
	n++;
	if (!testADMMGPUConst1Kappa()) return n;
	n++;
	if (!testADMMGPUConst1Bt1()) return n;
	n++;
	if (!testADMMGPUConst1CP()) return n;
	n++; 
	if (!testADMMGPUConstCpb()) return n;
	n++;
	if (!testADMMGPUConst1TradeP()) return n;
	n++; //10
	if (!testADMMGPUConst1Q()) return n;
	n++; 
	if (!testADMMGPUConst1alpha()) return n;
	n++;
	//std::cout << n << std::endl;
	if (!testADMMGPUConst1UpdateRes()) return n;
	n++;
	if (!testADMMGPUConst1CalcRes()) return n;
	n++;
	if (!testADMMGPUConst1Solve1()) return n;
	n++; 
	if (!testADMMGPUConst1Solve2()) return n;
	n++;
	if (!testADMMGPUConst1Solve3()) return n;
	n++;
	return 0;
}

void testADMMGPUConst1Time(int test) {

	switch (test)
	{
	case 0:
		testADMMGPUConst1TimeLAMBDA();
		break;
	case 1:
		testADMMGPUConst1TimeBt1();
		break;
	case 2:
		testADMMGPUConst1TimeTradeP();
		break;
	case 3:
		testADMMGPUConst1TimeUpdateRes();
		break;
	case 4:
		testADMMGPUConst1TimeCalcRes();
		break;
	default:
		std::cout << "No valid input " << std::endl;
		break;
	}
	
}

bool testADMMGPUConst1Contruct1()
{
	std::cout << "contructeur par defaut" << std::endl;
	ADMMGPUConst1 a;
	return true;
}

bool testADMMGPUConst1Contruct2()
{
	float rho = 2;

	std::cout << "contructeur avec parametres" << std::endl;
	ADMMGPUConst1 a(rho);
	return true;
}
bool testADMMGPUConst1Contruct3()
{
	float rho = 2;

	std::cout << "contructeur en deux temps" << std::endl;
	ADMMGPUConst1 a;
	a = ADMMGPUConst1(rho);
	return true;
}

bool testADMMGPUConst1Solve1()
{
	//solve(Simparam* result, Simparam sim, StudyCase cas);
	std::cout << "-------------------------------------------------------- " << std::endl;
	StudyCase cas;
	cas.Set2node();
	//cas.display();
	int nAgent = cas.getNagent();
	Simparam param(nAgent, 1);
	param.setRho(1);
	Simparam res(param);

	ADMMGPUConst1 a;

	a.solve(&res, param, cas);
	res.display();
	MatrixCPU Trade(nAgent, nAgent);
	Trade.set(0, 1, -1);
	Trade.set(1, 0, 1);
	MatrixCPU Res(res.getRes());
	Res.display();

	MatrixCPU trade = res.getTrade();
	trade.display();
	return trade.isEqual(&Trade, 0.001);

}
bool testADMMGPUConst1Solve2()
{
	//solve(Simparam* result, Simparam sim, StudyCase cas);
	std::cout << "-------------------------------------------------------- " << std::endl;
	StudyCase cas;
	cas.Set29node();
	//cas.display();
	int nAgent = cas.getNagent();

	Simparam param(nAgent, cas.getNLine());
	float epsG = 0.00002f;
	float epsL = 0.000002f;
	param.setEpsL(epsL);
	param.setEpsG(epsG);
	param.setStep(1, 1);
	param.setRho(10000);
	Simparam res(param);
	ADMMGPUConst1 a;
	a.solve(&res, param, cas);
	res.display();
	MatrixCPU Trade = res.getTrade();
	MatrixCPU P2(29, 1, 0);
	P2.sum(&Trade);

	float Pn[31] = { -1.008853555,-4.62966156,-2.927534103,-0.8979898691,-0.9462603927,-0.09805059433,-0.127968356,-4.168303013,-3.151874542,-2.261414766,-0.670329392,-3.399893284,-0.4841034412,-2.775528431,-3.008597374,-1.849177122,-0.5534118414,-2.362840891,-1.122991204,-0.1379692554,-2.332088947,4.406820297,5.406073093,3.676487684,3.929354668,4.570535183,2.529039145,3.478654861,2.755935192,3.768760443,4.393183708, };

	MatrixCPU P(31, 1);
	for (int i = 0; i < 31; i++) {
		P.set(i, 0, Pn[i]);
	}
	MatrixCPU P22 = res.getPn(); 
	

	return (P2.isEqual(&P, 0.01) && P2.isEqual(&P22, 0.01));

}

bool testADMMGPUConst1Solve3()
{
	std::cout << "-------------------------------------------------------- " << std::endl;
	StudyCase cas;
	float lim = 0.8;
	cas.Set2nodeConstraint(lim);
	int nAgent = cas.getNagent();
	Simparam param(nAgent, 1);
	Simparam res(param);
	
	param.setRho1(50);
	float value = (1 - lim) * (lim > 1) + lim;

	ADMMGPUConst1 a;

	MatrixCPU Trade(nAgent, nAgent);
	Trade.set(0, 1, -value);
	Trade.set(1, 0, value);
	a.solve(&res, param, cas);

	MatrixCPU trade = res.getTrade();
	res.display();
	trade.display();
	return trade.isEqual(&Trade, 0.001);
}

bool testADMMGPUConst1LAMBDA()
{
	int nAgent = 3; // 2 conso et un prod
	int ntrade = 4;
	int blockSize = 256;
	int numBlocks = ceil((nAgent + blockSize - 1) / blockSize);
	float value1 = 2;
	float value2 = -8;
	float value3 = 1.5;
	float value4 = 4;
	MatrixGPU LAMBDALin(ntrade, 1, value1);
	MatrixGPU trade(ntrade, 1, value2);
	MatrixGPU CoresLinTrans(ntrade, 1);
	
	MatrixGPU LAMBDALin2(ntrade, 1, value1 + 0.5 * value3 * (value2 + value4));
	float rho = value3;


	CoresLinTrans.set(0, 0, 2);
	CoresLinTrans.set(1, 0, 3);
	CoresLinTrans.set(2, 0, 0);
	CoresLinTrans.set(3, 0, 1);

	trade.set(2, 0, value4);
	trade.set(3, 0, value4);

	
	trade.transferGPU();
	LAMBDALin.transferGPU();
	CoresLinTrans.transferGPU();
	
	updateLAMBDAGPU << <numBlocks, blockSize >> > (LAMBDALin._matrixGPU, trade._matrixGPU, rho, CoresLinTrans._matrixGPU, ntrade);

	
	LAMBDALin.transferCPU();

	//LAMBDALin.display();
	//std::cout << "--------------" << std::endl;
	//LAMBDALin2.display();


	return (LAMBDALin.isEqual(&LAMBDALin2));
}

bool testADMMGPUConst1Kappa()
{
	
	int _nLine = 3;
	
	float value1 = 2;
	float value2 = 3;
	float value3 = 1;
	float value4 = -2;
	
	int _blockSize = 256;
	int _numBlocksL = ceil((_nLine + _blockSize - 1) / _blockSize);

	
	
	
	MatrixCPU Qtot(_nLine, 1, value1);
	MatrixCPU Llimit(_nLine, 1, value2);
	MatrixCPU Kappa1(_nLine, 1, value3);
	MatrixCPU Kappa2(_nLine, 1, value4);
	
	Kappa1.projectNeg();
	Kappa1.add(&Llimit);
	Kappa1.subtract(&Qtot);
	
	Kappa2.projectNeg();
	Kappa2.add(&Llimit);
	Kappa2.add(&Qtot);


	MatrixGPU QtotGPU(_nLine, 1, value1, 1);
	MatrixGPU LlimitGPU(_nLine, 1, value2, 1);
	MatrixGPU Kappa1GPU(_nLine, 1, value3, 1);
	MatrixGPU Kappa2GPU(_nLine, 1, value4, 1);

	MatrixCPU Kappa1Result(_nLine, 1);
	MatrixCPU Kappa2Result(_nLine, 1);

	updateKappaGPU << <_numBlocksL, _blockSize >> > (Kappa1GPU._matrixGPU, Kappa2GPU._matrixGPU, LlimitGPU._matrixGPU, QtotGPU._matrixGPU, _nLine);

	Kappa1GPU.toMatCPU(Kappa1Result);
	Kappa2GPU.toMatCPU(Kappa2Result);

	

	return (Kappa1.isEqual(&Kappa1Result) && Kappa2.isEqual(&Kappa2Result));
}

bool testADMMGPUConst1Bt1()
{
	int nAgent = 3; // 2 conso et un prod
	int ntrade = 4;
	int blockSize = 256;
	int numBlocks = ceil((nAgent + blockSize - 1) / blockSize);
	float value1 = 2;
	float value2 = -8;
	float value3 = 1.5;
	float value4 = 4;
	MatrixGPU Bt1(ntrade, 1, 0);
	MatrixGPU Bt11(ntrade, 1, -value4 - value1 / value3);
	MatrixGPU trade(ntrade, 1, value2);
	MatrixGPU CoresLinTrans(ntrade, 1);

	MatrixGPU LAMBDALin(ntrade, 1, value1 + 0.5 * value3 * (value2 + value4));
	float rho = value3;


	CoresLinTrans.set(0, 0, 2);
	CoresLinTrans.set(1, 0, 3);
	CoresLinTrans.set(2, 0, 0);
	CoresLinTrans.set(3, 0, 1);

	trade.set(2, 0, value4);
	trade.set(3, 0, value4);
	Bt11.set(2, 0, -value2 - value1 / value3);
	Bt11.set(3, 0, -value2 - value1 / value3);



	Bt1.transferGPU();
	trade.transferGPU();
	LAMBDALin.transferGPU();

	CoresLinTrans.transferGPU();



	updateBt1GPU << <numBlocks, blockSize >> > (Bt1._matrixGPU, trade._matrixGPU, rho, LAMBDALin._matrixGPU, CoresLinTrans._matrixGPU, ntrade);

	Bt1.transferCPU();
	LAMBDALin.transferCPU();

	return Bt1.isEqual(&Bt11);
}

bool testADMMGPUConst1TradeP()
{
	int nAgent = 4;
	int ntrade = 8;
	int blockSize = 256;
	//int numBlocks = ceil((nAgent + blockSize - 1) / blockSize);
	float value1 = 2;
	float value2 = 3;
	float value3 = 1;
	float value4 = 1;
	float value5 = -1;
	float value6 = -30;
	float value7 = 10;
	float value8 = 5;
	float value9 = value8 - value3 + value4 - value5;
	MatrixGPU Bt1(ntrade, 1, value1);
	float at1 = value3;
	float at2 = value4;
	MatrixGPU Ct(ntrade, 1, value5);
	MatrixGPU Lb(ntrade, 1, value6);
	MatrixGPU Ub(ntrade, 1, value7);
	MatrixGPU Tlocal(ntrade, 1);
	
	MatrixGPU Tlocal_pre(ntrade, 1, value8);
	//Tlocal_pre.set(0,0,value8+1);
	MatrixGPU Tlocal2(Tlocal_pre);
	MatrixGPU CoresLinAgent(ntrade, 1);
	CoresLinAgent.set(1, 0, 0);
	CoresLinAgent.set(2, 0, 1);
	CoresLinAgent.set(3, 0, 1);
	CoresLinAgent.set(4, 0, 2);
	CoresLinAgent.set(5, 0, 2);
	CoresLinAgent.set(6, 0, 3);
	CoresLinAgent.set(7, 0, 3);
	MatrixGPU Bp1(nAgent, 1);
	MatrixGPU Bp11(nAgent, 1);
	MatrixGPU Ap1(nAgent, 1, value2);
	MatrixGPU Ap2(nAgent, 1, value3);
	MatrixGPU Ap12(nAgent, 1, value2 + value3);
	MatrixGPU Cp(nAgent, 1, value5);
	MatrixGPU PLb(nAgent, 1, value6/2);
	MatrixGPU PUb(nAgent, 1, value7/2);

	MatrixGPU nVoisin(nAgent, 1, 2);
	MatrixGPU Tmoy(nAgent, 1, value3);
	MatrixGPU Tmoy2(Tmoy);
	MatrixGPU MU(nAgent, 1, value5);
	MatrixGPU MU2(MU);
	MatrixGPU P(nAgent, 1, value4);
	MatrixGPU P2(P);

	MatrixGPU CoresAgentLin(nAgent + 1, 1);
	CoresAgentLin.set(1, 0, 2);
	CoresAgentLin.set(2, 0, 4);
	CoresAgentLin.set(3, 0, 6);
	CoresAgentLin.set(4, 0, 8);

	
	for (int i = 0; i < nAgent; i += 1) // 1 bloc = 1 agent
	{
		float s = 0.0;
		for (int j = CoresAgentLin.get(i, 0); j < CoresAgentLin.get(i + 1, 0); j++) // on parcourt les trades de l'agent i
		{
			float m = Tlocal2.get(j, 0) - Tmoy2.get(i, 0) + P2.get(i, 0) - MU2.get(i, 0);
			float r = (Bt1.get(j, 0) * at1 + m * at2 - Ct.get(j, 0)) / (at1 + at2);
			float ub = Ub.get(j, 0);
			float lb = Lb.get(j, 0);
			float t = (ub - r) * (r > ub) + (lb - r) * (r < lb) + r;
			Tlocal2.set(j, 0, t);
			s += t;
		}
		float r = s / nVoisin.get(i, 0);
		Tmoy2.set(i, 0, r);
		Bp11.set(i, 0, r + MU2.get(i, 0));
		float p = (Ap1.get(i, 0) * Bp11.get(i, 0) - Cp.get(i, 0)) / (Ap12.get(i, 0));
		float ub = PUb.get(i, 0);
		float lb = PLb.get(i, 0);
		p = (ub - p) * (p > ub) + (lb - p) * (p < lb) + p;
		P2.set(i, 0, p);
		MU2.set(i, 0, MU2.get(i, 0) + r - P2.get(i, 0));
	}
	
	


	Bt1.transferGPU();
	Tlocal.transferGPU();
	Ct.transferGPU();
	Lb.transferGPU();
	Ub.transferGPU();
	Tlocal_pre.transferGPU();
	Tmoy.transferGPU();
	P.transferGPU();
	MU.transferGPU();
	CoresLinAgent.transferGPU();
	Ap1.transferGPU();
	Ap2.transferGPU();
	Ap12.transferGPU();
	Bp1.transferGPU();
	Cp.transferGPU();
	PLb.transferGPU();
	PUb.transferGPU();
	nVoisin.transferGPU();
	CoresAgentLin.transferGPU();

	std::cout << " fin transfert " << std::endl;


	updateTradePGPU<256> <<<nAgent, blockSize >> > (Tlocal._matrixGPU, Tlocal_pre._matrixGPU, Tmoy._matrixGPU, P._matrixGPU, MU._matrixGPU, nVoisin._matrixGPU, at1, at2, Bt1._matrixGPU, Ct._matrixGPU,
		Lb._matrixGPU, Ub._matrixGPU, Ap1._matrixGPU, Ap12._matrixGPU, Cp._matrixGPU, PLb._matrixGPU, PUb._matrixGPU, CoresAgentLin._matrixGPU, CoresLinAgent._matrixGPU, nAgent);

	Tlocal.transferCPU();
	P.transferCPU();
	MU.transferCPU();
	Tmoy.transferCPU();
	
	Tlocal2.display();
	Tlocal.display();
	Tmoy2.display();
	Tmoy.display();
	P2.display();
	P.display();
	MU2.display();
	MU.display();

	Bp11.display();


	return ((Tlocal.isEqual(&Tlocal2)) && (P.isEqual(&P2)) && (MU.isEqual(&MU2)) && (Tmoy.isEqual(&Tmoy2)));
}

bool testADMMGPUConst1alpha()
{
	int _nAgent = 2;
	int _nLine = 3;

	float value1 = 2;
	float value2 = 3;
	
	int _blockSize = 256;
	int _numBlocksNL = ceil((_nLine*_nAgent + _blockSize - 1) / _blockSize);


	MatrixCPU Pn(_nAgent, 1, value1);
	MatrixCPU G(_nLine, _nAgent, value2);
	MatrixCPU alpha(_nLine, _nAgent);
	
	alpha.multiplyTVector(&G, &Pn, 0);
	
	MatrixGPU PnGPU(_nAgent, 1, value1, 1);
	MatrixGPU GGPU(_nLine, _nAgent, value2, 1);
	MatrixGPU alphaGPU(_nLine, _nAgent, 0, 1);

	updateAlpha << < _numBlocksNL, _blockSize >> > (alphaGPU._matrixGPU, GGPU._matrixGPU, PnGPU._matrixGPU, _nLine, _nAgent);

	MatrixCPU alphaResult(_nLine, _nAgent);
	
	alphaGPU.toMatCPU(alphaResult);
	


	return alpha.isEqual(&alphaResult);
}

bool testADMMGPUConst1Q()
{
	int _nAgent = 10;
	int _nLine = 7;

	float value1 = 2;
	
	int _blockSize = 256;
	int _numBlocksL = ceil((_nLine + _blockSize - 1) / _blockSize);

	MatrixCPU alpha(_nLine, _nAgent, value1);
	MatrixCPU Qpart(_nLine, _nAgent);
	MatrixCPU Qtot(_nLine, 1);
	

	for (int l = 0; l < _nLine; l++) {
		float qt = 0;
		for (int n = _nAgent - 1; n >= 0; n--) {
			qt += alpha.get(l, n);
			if (n > 0) {
				Qpart.set(l, n - 1, qt);
			}
		}
		Qtot.set(l, 0, qt);
	}

	MatrixGPU alphaGPU(_nLine, _nAgent, value1, 1);
	MatrixGPU QpartGPU(_nLine, _nAgent, 0, 1);
	MatrixGPU alphaGPUTrans(_nAgent, _nLine, value1, 1);
	MatrixGPU QpartGPUTrans(_nAgent, _nLine, 0, 1);
	MatrixGPU QtotGPU(_nLine, 1, 0, 1);
	MatrixGPU QtotGPU1(_nLine, 1, 0, 1);
	MatrixGPU QtotGPU2(_nLine, 1, 0, 1);

	
	updateQpart <<< _nLine, _blockSize, _nAgent * sizeof(float) >> > (QpartGPU._matrixGPU, alphaGPU._matrixGPU, _nAgent);
	calculQpartAgentBlocTrans <<< _nAgent, _blockSize >>> (QpartGPUTrans._matrixGPU, alphaGPUTrans._matrixGPU, _nLine, _nAgent);

	QtotGPU.sum(&alphaGPU);
	updateQtotTest <<< _numBlocksL, _blockSize >> > (QtotGPU1._matrixGPU, QpartGPU._matrixGPU, alphaGPU._matrixGPU, _nLine, _nAgent);
	updateQtotTestTrans << < _numBlocksL, _blockSize >> > (QtotGPU2._matrixGPU, QpartGPUTrans._matrixGPU, alphaGPU._matrixGPU, _nLine);

	MatrixCPU QpartResult(_nLine, _nAgent);
	MatrixCPU QpartResultTrans(_nAgent, _nLine);
	MatrixCPU QtotResult(_nLine, 1);
	MatrixCPU QtotResult1(_nLine, 1);
	MatrixCPU QtotResult2(_nLine, 1);
	
	QtotGPU.toMatCPU(QtotResult);
	QtotGPU1.toMatCPU(QtotResult1);
	QtotGPU2.toMatCPU(QtotResult2);
	QpartGPU.toMatCPU(QpartResult);
	QpartGPUTrans.toMatCPU(QpartResultTrans);

	std::cout << "-------------------------------------------------------- " << std::endl;
	alpha.display();
	QpartResult.display();
	QpartResultTrans.display();

	std::cout << "-------------------------------------------------------- " << std::endl;
	QtotResult.display();
	QtotResult1.display();
	QtotResult2.display();

	std::cout << (Qtot.isEqual(&QtotResult)) <<" " << (Qtot.isEqual(&QtotResult1)) << " " << (Qtot.isEqual(&QtotResult2)) << " " << (Qpart.isEqual(&QpartResult)) << std::endl;

	return (Qtot.isEqual(&QtotResult)) && (Qtot.isEqual(&QtotResult1)) && (Qtot.isEqual(&QtotResult2)) && (Qpart.isEqual(&QpartResult));
}

bool testADMMGPUConst1CP()
{
	int _nAgent = 2;
	int _nLine = 3;
	float _rho1 = 1.5;
	float value1 = 2;
	float value2 = 3;
	float value3 = 1;
	float value4 = -2;
	float value5 = -1;
	float value6 = -30;
	//float value7 = 10;
	//float value8 = 5;
	int _blockSize = 256;
	int numBlocks = _nAgent;
	int _numBlocksN = ceil((_nAgent + _blockSize - 1) / _blockSize);
	int _numBlocksL = ceil((_nLine + _blockSize - 1) / _blockSize);
	
	MatrixCPU Cp(_nAgent, 1);
	MatrixCPU tempN1(_nAgent, 1);
	MatrixCPU Cp1(_nAgent, 1, value1);
	MatrixCPU Cp2(_nAgent, 1, value2);
	MatrixCPU tempL1(_nLine, 1);
	MatrixCPU Kappa1(_nLine, 1, value3);
	MatrixCPU Kappa2(_nLine, 1, value4);
	MatrixCPU G(_nLine, _nAgent, value5);
	MatrixCPU Qpart(_nLine, _nAgent, value6);
	MatrixCPU nVoisin(_nAgent, 1, 1);


	ADMMGPUConst1 a;
	tempL1.subtractAbs(&Kappa1, &Kappa2);
	//Cp2->multiplyTrans(G, tempL1, 0);

	float r = 0;
	for (int i = 0; i < _nAgent; i++)
	{
		r = 0;
		for (int k = 0; k < _nLine; ++k)
		{
			r +=  G.get(k, i) * (tempL1.get(k, 0) + 2 * Qpart.get(k, i));
		}
		Cp2.set(i, 0, r);
	}

	Cp2.multiply(_rho1);
	Cp2.multiplyT(&nVoisin);

	Cp.add(&Cp1, &Cp2);
	

	MatrixGPU CpGPU(_nAgent, 1, 0, 1);
	MatrixGPU tempN1GPU(_nAgent, 1, 0, 1);
	MatrixGPU Cp1GPU(_nAgent, 1, value1, 1);
	MatrixGPU Cp2GPU(_nAgent, 1, value2, 1);
	MatrixGPU tempL1GPU(_nLine, 1, 0, 1);
	MatrixGPU Kappa1GPU(_nLine, 1, value3, 1);
	MatrixGPU Kappa2GPU(_nLine, 1, value4, 1);
	MatrixGPU GGPU(_nLine, _nAgent, value5, 1);
	MatrixGPU QpartGPU(_nLine, _nAgent, value6, 1);
	MatrixCPU CpResult(_nAgent, 1);
	MatrixGPU nVoisinGPU(_nAgent, 1, 1, 1);

	diffKappa << <_numBlocksL, _blockSize >> > (tempL1GPU._matrixGPU, Kappa1GPU._matrixGPU, Kappa2GPU._matrixGPU, _nLine);
	updateCp2a<256> << <numBlocks, _blockSize >> > (Cp2GPU._matrixGPU, tempL1GPU._matrixGPU, GGPU._matrixGPU, _nLine, _nAgent);
	updateCp2b<256> << <numBlocks, _blockSize >> > (tempN1GPU._matrixGPU, GGPU._matrixGPU, QpartGPU._matrixGPU, _nLine, _nAgent);
	updateCpOld << <_numBlocksN, _blockSize >> > (CpGPU._matrixGPU, Cp1GPU._matrixGPU, Cp2GPU._matrixGPU, tempN1GPU._matrixGPU, nVoisinGPU._matrixGPU, _rho1, _nAgent);

	CpGPU.toMatCPU(CpResult);

	Cp.display();
	CpResult.display();

	return Cp.isEqual(&CpResult);
}

bool testADMMGPUConstCpb()
{
	int _nAgent = 100;
	int _nLine = 200;

	float value1 = 2;
	float value2 = 3;
	int _blockSize = 256;
	int numBlocks = _nAgent;

	
	MatrixCPU Qpart(_nLine, _nAgent, value1);
	MatrixCPU G(_nLine, _nAgent, value2);
	MatrixCPU Cpb(_nAgent, 1);

	for (int n = 0; n < _nAgent; n++) {
		float sum = 0;
		for (int l = 0; l < _nLine; l++) {
			sum += G.get(l, n) * Qpart.get(l, n);
		}
		Cpb.set(n, 0, 2*sum);
	}
	

	MatrixGPU GGPU(_nLine, _nAgent, value2, 1);
	MatrixGPU GGPUTrans(_nAgent, _nLine, value2, 1);
	MatrixGPU QpartGPU(_nLine, _nAgent, value1, 1);
	MatrixGPU QpartGPUTrans(_nAgent, _nLine, value1, 1);

	MatrixGPU CpbGPU(_nAgent, 1, 0, 1);
	MatrixGPU CpbGPU1(_nAgent, 1, 0, 1);



	
	updateCp2bTest<256> << <numBlocks, _blockSize >> > (CpbGPU._matrixGPU, GGPU._matrixGPU, QpartGPU._matrixGPU, _nLine, _nAgent);
	updateCp2bTestTrans<256> << <numBlocks, _blockSize >> > (CpbGPU1._matrixGPU, GGPUTrans._matrixGPU, QpartGPUTrans._matrixGPU, _nLine, _nAgent);

	MatrixCPU CpbResult(_nAgent, 1);
	MatrixCPU CpbResult2(_nAgent, 1);


	CpbGPU.toMatCPU(CpbResult);
	CpbGPU1.toMatCPU(CpbResult2);
	

	
	return (Cpb.isEqual(&CpbResult)) && (Cpb.isEqual(&CpbResult2)) ;

}

bool testADMMGPUConst1UpdateRes()
{	/*float ADMMGPU5::updateRes(MatrixCPU* res, MatrixGPU* Tlocal, MatrixGPU* trade, int iter, MatrixGPU* CoresLinAgent, MatrixGPU* CoresLinVoisin, MatrixGPU* CoresMatLin, MatrixGPU* tempNN)
{
	tempNN->subtract(Tlocal, trade);
	//hipDeviceSynchronize();
	float resS = tempNN->distance2();

	updateDiffGPU <<<_numBlocks, _blockSize >>> (tempNN->_matrixGPU, Tlocal->_matrixGPU, CoresLinTrans->_matrixGPU, _N);
	//hipDeviceSynchronize();
	float resR = tempNN->distance2();

	res->set(0, iter, resR);
	res->set(1, iter, resS);


	return resR * (resR > resS) + resS * (resR <= resS);*/
	int ntrade = 4;
	int blockSize = 15;
	int numBlocks = ceil((ntrade + blockSize - 1) / blockSize);
	std::cout << "blockSize " << blockSize << " numBlocks " << numBlocks << std::endl;
	float value1 = 4;
	float value2 = 2.5;
	float value3 = -2;
	float value4 = value3 + value2 - value1;
	MatrixCPU res(2, 1);
	MatrixCPU res2(2, 1);
	MatrixGPU Tlocal(ntrade, 1, value1);
	MatrixGPU Tlocal_pre(ntrade, 1, value2);

	MatrixGPU CoresLinTrans(ntrade, 1);

	MatrixGPU tempN(numBlocks, 1, 0, 1);
	MatrixGPU tempN2(numBlocks, 1, 0, 1);
	
	

	CoresLinTrans.set(0, 0, 2);
	CoresLinTrans.set(1, 0, 3);
	CoresLinTrans.set(2, 0, 0);
	CoresLinTrans.set(3, 0, 1);

	Tlocal.set(2, 0, value3);
	Tlocal.set(3, 0, value3);
	Tlocal_pre.set(2, 0, value4);
	Tlocal_pre.set(3, 0, value4);



	res2.set(0, 0, sqrtf((value1 + value3) * (value1 + value3) ));
	res2.set(1, 0, sqrtf((value1 - value2) * (value1 - value2) ));
	
	Tlocal.transferGPU();
	Tlocal_pre.transferGPU();
	CoresLinTrans.transferGPU();
	
	float resS = Tlocal.max2(&Tlocal_pre);
	updateDiffGPU << <numBlocks, blockSize >> > (tempN._matrixGPU, Tlocal._matrixGPU, CoresLinTrans._matrixGPU, ntrade);
	float resR = tempN.max2();
	
	res.set(0, 0, resR);
	res.set(1, 0, resS);

	return res2.isEqual(&res);
}
bool testADMMGPUConst1CalcRes() {
	/*
	* float ADMMGPU5::calcRes( MatrixGPU* Tlocal, MatrixGPU* P, MatrixGPU* tempN1, MatrixGPU* tempNN)
{
	 tempNN->subtract(Tlocal, &Tlocal_pre);
	 tempN1->subtract(&Tmoy, P);

	 float d1 = tempN1->max2();
	 float d2 = tempNN->max2();


	 return d1* (d1 > d2) + d2 * (d2 >= d1);
}*/

	int nAgent = 3;
	int ntrade = 4;
	int blockSize = 256;
	int numBlocks = ceil((ntrade + blockSize - 1) / blockSize);
	float value1 = 5;
	float value2 = 2;
	float value3 = -3;
	float value4 = -1;
	MatrixGPU Tlocal(ntrade, 1, value1);
	MatrixGPU Tlocal_pre(ntrade, 1, value2);
	MatrixGPU Tmoy(nAgent, 1, value3);
	MatrixGPU P(nAgent, 1, value4);


	Tlocal.transferGPU();
	Tlocal_pre.transferGPU();
	Tmoy.transferGPU();
	P.transferGPU();
	

	float d11 = Tlocal.max2(&Tlocal_pre);
	float d22 = P.max2(&Tmoy);
	float d = d11 * (d11 > d22) + d22 * (d22 >= d11);



	float d1 = fabs(value1 - value2);
	float d2 = fabs(value3 - value4);
	float df = d1 * (d1 > d2) + d2 * (d2 >= d1);



	return (df == d);
}


void testADMMGPUConst1TimeLAMBDA()
{
	std::string fileName = "TempsLAMBDA.csv";
	//hipEvent_t start, stop;
	//float elapsedTime;
	std::chrono::high_resolution_clock::time_point a;
	std::chrono::high_resolution_clock::time_point b;
	unsigned int time;
	std::ios_base::openmode mode = std::fstream::in | std::fstream::out | std::fstream::app;
	const int nNAgent = 7;
	const int nSimu = 100;
	const int nRepet = 10;
	int nAgent[nNAgent] = { 10, 100, 500, 1000, 5000, 10000, 40000 }; // autant conso que de prod, la derni�re veleur ne "marche" pas (trop rapide)
	int ntrade[nNAgent];
	int blockSize = 256;
	float values1[nSimu];
	float values2[nSimu];
	float rhos[nSimu];
	MatrixCPU temps(nNAgent, nSimu, 0);
	
	for (int j = 0; j < nSimu;j++) {
		values1[j] = (float) (rand()) / rand();
		values2[j] = (float)(rand()) / rand();
		rhos[j] = (float)(rand()%100) / rand();
	}

	for (int i = 0; i < nNAgent; i++) {
		
		ntrade[i] = nAgent[i] * nAgent[i] / 2;
		std::cout << "iteration " << i << " nAgent " << nAgent[i] << " ntrade " << ntrade[i] << std::endl;
		int numBlocks = ceil((ntrade[i] + blockSize - 1) / blockSize);
		
		MatrixGPU CoresLinAgent(ntrade[i], 1);
		MatrixGPU CoresLinVoisin(ntrade[i], 1);
		MatrixGPU CoresMatLin(nAgent[i], nAgent[i], -1);
		MatrixGPU CoresLinTrans(ntrade[i], 1);
		

		int indice = 0;
		int idVoisin = 0;

		for (int idAgent = 0; idAgent < nAgent[i]; idAgent++) {
			int Nvoisinmax = nAgent[i] / 2;
			if (idAgent < Nvoisinmax) {
				idVoisin = 0;
			}
			else {
				idVoisin = nAgent[i] / 2;
			}
			for (int voisin = idVoisin; voisin < Nvoisinmax; voisin++) {
				CoresLinAgent.set(indice, 0, idAgent);
				CoresLinVoisin.set(indice, 0, voisin);
				CoresMatLin.set(idAgent, voisin, indice);
				indice = indice + 1;
			}
		}
		for (int lin = 0;lin < ntrade[i];lin++) {
			int i = CoresLinAgent.get(lin, 0);
			int j = CoresLinVoisin.get(lin, 0);
			int k = CoresMatLin.get(j, i);
			CoresLinTrans.set(lin, 0, k);
		}
		CoresLinAgent.transferGPU();
		CoresLinVoisin.transferGPU();
		CoresMatLin.transferGPU();
		CoresLinTrans.transferGPU();
		clock_t t = clock();
		
		for (int simu = 0; simu < nSimu; simu++) {
			MatrixGPU LAMBDALin(ntrade[i], 1, values1[simu], 1);
			MatrixGPU trade(ntrade[i], 1, values2[simu], 1);
			float rho = rhos[simu];
			/*hipEventCreate(&start);
			hipEventRecord(start, 0);*/
			hipDeviceSynchronize();
			time = 0;
			for (int repet = 0; repet < nRepet; repet++) {
				MatrixGPU LAMBDALinCopy(LAMBDALin);
				MatrixGPU tradeCopy(trade);
				MatrixGPU CoresLinTransCopy(CoresLinTrans);
				hipDeviceSynchronize();
				a = std::chrono::high_resolution_clock::now();
				updateLAMBDAGPU <<<numBlocks, blockSize >>> (LAMBDALinCopy._matrixGPU, tradeCopy._matrixGPU, rho, CoresLinTransCopy._matrixGPU, ntrade[i]);
				hipDeviceSynchronize();
				b = std::chrono::high_resolution_clock::now();
				time += std::chrono::duration_cast<std::chrono::nanoseconds>(b - a).count();
			}
			temps.set(i, simu, (float) time/nRepet);
		}
	}
	temps.saveCSV(fileName, mode);
}

void testADMMGPUConst1TimeBt1()
{
	std::string fileName = "TempsBt.csv";
	//hipEvent_t start, stop;
	//float elapsedTime;
	std::chrono::high_resolution_clock::time_point a;
	std::chrono::high_resolution_clock::time_point b;
	unsigned int time;
	std::ios_base::openmode mode = std::fstream::in | std::fstream::out | std::fstream::app;
	const int nNAgent = 7;
	const int nSimu = 100;
	const int nRepet = 10;
	int nAgent[nNAgent] = { 10, 100, 500, 1000, 5000, 10000, 40000 }; // autant conso que de prod, la derni�re veleur ne "marche" pas (trop rapide)
	int ntrade[nNAgent];
	int blockSize = 256;
	float values1[nSimu];
	float values2[nSimu];
	float rhos[nSimu];
	MatrixCPU temps(nNAgent, nSimu, 0);

	for (int j = 0; j < nSimu;j++) {
		values1[j] = (float)(rand()) / rand();
		values2[j] = (float)(rand()) / rand();
		rhos[j] = (float)(rand() % 100) / rand();
	}

	for (int i = 0; i < nNAgent; i++) {

		ntrade[i] = nAgent[i] * nAgent[i] / 2;
		std::cout << "iteration " << i << " nAgent " << nAgent[i] << " ntrade " << ntrade[i] << std::endl;
		int numBlocks = ceil((ntrade[i] + blockSize - 1) / blockSize);

		MatrixGPU Bt1(ntrade[i], 1, 0, 1);

		MatrixGPU CoresLinAgent(ntrade[i], 1);
		MatrixGPU CoresLinVoisin(ntrade[i], 1);
		MatrixGPU CoresMatLin(nAgent[i], nAgent[i], -1);
		MatrixGPU CoresLinTrans(ntrade[i], 1);


		int indice = 0;
		int idVoisin = 0;

		for (int idAgent = 0; idAgent < nAgent[i]; idAgent++) {
			int Nvoisinmax = nAgent[i] / 2;
			if (idAgent < Nvoisinmax) {
				idVoisin = 0;
			}
			else {
				idVoisin = nAgent[i] / 2;
			}
			for (int voisin = idVoisin; voisin < Nvoisinmax; voisin++) {
				CoresLinAgent.set(indice, 0, idAgent);
				CoresLinVoisin.set(indice, 0, voisin);
				CoresMatLin.set(idAgent, voisin, indice);
				indice = indice + 1;
			}
		}
		for (int lin = 0;lin < ntrade[i];lin++) {
			int i = CoresLinAgent.get(lin, 0);
			int j = CoresLinVoisin.get(lin, 0);
			int k = CoresMatLin.get(j, i);
			CoresLinTrans.set(lin, 0, k);
		}
		CoresLinAgent.transferGPU();
		CoresLinVoisin.transferGPU();
		CoresMatLin.transferGPU();
		CoresLinTrans.transferGPU();
		clock_t t = clock();

		for (int simu = 0; simu < nSimu; simu++) {
			MatrixGPU LAMBDALin(ntrade[i], 1, values1[simu], 1);
			MatrixGPU trade(ntrade[i], 1, values2[simu], 1);
			float rho = rhos[simu];
			/*hipEventCreate(&start);
			hipEventRecord(start, 0);*/
			hipDeviceSynchronize();
			time = 0;
			for (int repet = 0; repet < nRepet; repet++) {
				MatrixGPU LAMBDALinCopy(LAMBDALin);
				MatrixGPU tradeCopy(trade);
				MatrixGPU CoresLinTransCopy(CoresLinTrans);
				MatrixGPU Bt1Copy(Bt1);
				hipDeviceSynchronize();
				a = std::chrono::high_resolution_clock::now();
				updateBt1GPU << <numBlocks, blockSize >> > (Bt1Copy._matrixGPU, tradeCopy._matrixGPU, rho, LAMBDALinCopy._matrixGPU, CoresLinTransCopy._matrixGPU, ntrade[i]);
				hipDeviceSynchronize();
				b = std::chrono::high_resolution_clock::now();
				time += std::chrono::duration_cast<std::chrono::nanoseconds>(b - a).count();
			}
			temps.set(i, simu, (float)time / nRepet);
		}
	}
	temps.saveCSV(fileName, mode);
}


void testADMMGPUConst1TimeTradeP() {
	std::string fileName = "TempsTradeP8.csv";
	std::chrono::high_resolution_clock::time_point a;
	std::chrono::high_resolution_clock::time_point b;
	unsigned int time;
	std::ios_base::openmode mode = std::fstream::in | std::fstream::out | std::fstream::app;
	const int nNAgent = 6;
	const int nSimu = 100;
	const int nRepet = 10;
	const int nVar = 14;
	int nAgent[nNAgent] = { 10, 100, 500, 1000, 5000, 10000 }; // autant conso que de prod, 
	int ntrade[nNAgent];
	int blockSize = 256;
	float values[nVar][nSimu];
	float rhos[nSimu];
	MatrixCPU temps(nNAgent, nSimu, 0);

	for (int j = 0; j < nSimu; j++) {
		for (int var = 0; var < nVar; var++) {
			values[var][j] = (float)(rand()) / rand();
		}
		rhos[j] = (float)(rand() % 100) / rand();
	}

	for (int i = 0; i < nNAgent; i++) {

		ntrade[i] = nAgent[i] * nAgent[i] / 2;
		std::cout << "iteration " << i << " nAgent " << nAgent[i] << " ntrade " << ntrade[i] << std::endl;
	
		MatrixGPU Tlocal(ntrade[i], 1, 0, 1);
		MatrixGPU CoresLinVoisin(ntrade[i], 1);
		MatrixGPU CoresAgentLin(nAgent[i] + 1, 1);

		int indice = 0;
		int idVoisin = 0;

		for (int idAgent = 0; idAgent < nAgent[i]; idAgent++) {
			int Nvoisinmax = nAgent[i] / 2;
			if (idAgent < Nvoisinmax) {
				idVoisin = 0;
			}
			else {
				idVoisin = nAgent[i] / 2;
			}
			for (int voisin = idVoisin; voisin < Nvoisinmax; voisin++) {
				CoresLinVoisin.set(indice, 0, voisin);

				indice = indice + 1;
			}
			CoresAgentLin.set(idAgent + 1, 0, indice);


		}
		CoresLinVoisin.transferGPU();
		CoresAgentLin.transferGPU();
		clock_t t = clock();

		for (int simu = 0; simu < nSimu; simu++) {

			MatrixGPU Tlocal_pre(ntrade[i], 1, values[0][simu], 1);
			MatrixGPU Bt1(ntrade[i], 1, values[1][simu], 1);
			MatrixGPU Ct(ntrade[i], 1, values[2][simu], 1);
			MatrixGPU Lb(ntrade[i], 1, values[3][simu], 1);
			MatrixGPU Ub(ntrade[i], 1, values[4][simu], 1);

			MatrixGPU Tmoy(nAgent[i], 1, values[5][simu], 1);
			MatrixGPU P(nAgent[i], 1, values[6][simu], 1);
			MatrixGPU MU(nAgent[i], 1, values[7][simu], 1);
			MatrixGPU nVoisin(nAgent[i], 1, nAgent[i] / 2, 1);
			MatrixGPU Ap1(nAgent[i], 1, values[8][simu], 1);
			MatrixGPU Ap12(nAgent[i], 1, values[10][simu], 1);
			MatrixGPU Cp(nAgent[i], 1, values[11][simu], 1);
			MatrixGPU PLb(nAgent[i], 1, values[12][simu], 1);
			MatrixGPU PUb(nAgent[i], 1, values[13][simu], 1);

			float rho = rhos[simu];
			float at1 = 2 * rho;
			float at2 = 3 * rho;


			time = 0;
			for (int repet = 0; repet < nRepet; repet++) {
				int _n = nAgent[i];
				MatrixGPU Tlocal_preCopy(Tlocal_pre);
				MatrixGPU Bt1Copy(Bt1);
				MatrixGPU CtCopy(Ct);
				MatrixGPU LbCopy(Lb);
				MatrixGPU UbCopy(Ub);

				MatrixGPU TmoyCopy(Tmoy);
				MatrixGPU PCopy(P);
				MatrixGPU MUCopy(MU);
				MatrixGPU nVoisinCopy(nVoisin);
				MatrixGPU Ap1Copy(Ap1);
				MatrixGPU Ap12Copy(Ap12);
				MatrixGPU CpCopy(Cp);
				MatrixGPU PLbCopy(PLb);
				MatrixGPU PUbCopy(PUb);

				MatrixGPU TlocalCopy(Tlocal);
				MatrixGPU CoresLinVoisinCopy(CoresLinVoisin);
				MatrixGPU CoresAgentLinCopy(CoresAgentLin);

				hipDeviceSynchronize();
				a = std::chrono::high_resolution_clock::now();
				updateTradePGPU<256> << <_n, blockSize >> > (TlocalCopy._matrixGPU, Tlocal_preCopy._matrixGPU, TmoyCopy._matrixGPU, PCopy._matrixGPU,
					MUCopy._matrixGPU, nVoisinCopy._matrixGPU, at1, at2, Bt1Copy._matrixGPU, CtCopy._matrixGPU, LbCopy._matrixGPU, UbCopy._matrixGPU,
					Ap1Copy._matrixGPU, Ap12Copy._matrixGPU, CpCopy._matrixGPU, PLbCopy._matrixGPU, PUbCopy._matrixGPU, CoresAgentLinCopy._matrixGPU, CoresLinVoisinCopy._matrixGPU, _n);
				hipDeviceSynchronize();
				b = std::chrono::high_resolution_clock::now();
				time += std::chrono::duration_cast<std::chrono::nanoseconds>(b - a).count();
			}
			temps.set(i, simu, (float)time / nRepet);
		}
	}

	temps.saveCSV(fileName, mode);
}



void testADMMGPUConst1TimeUpdateRes() {
	std::string fileName = "TempsResG8.csv";
	
	std::chrono::high_resolution_clock::time_point a;
	std::chrono::high_resolution_clock::time_point b;
	unsigned int time;
	std::ios_base::openmode mode = std::fstream::in | std::fstream::out | std::fstream::app;
	const int nNAgent = 7;
	const int nSimu = 100;
	const int nRepet = 10;
	int nAgent[nNAgent] = { 10, 100, 500, 1000, 5000, 10000, 40000 }; // autant conso que de prod, la derni�re veleur ne "marche" pas (trop rapide)
	int ntrade[nNAgent];
	int blockSize = 256;
	float values1[nSimu];
	float values2[nSimu];
	
	MatrixCPU temps(nNAgent, nSimu, 0);
	MatrixCPU res(2, 1, 0);

	for (int j = 0; j < nSimu;j++) {
		values1[j] = (float)(rand()) / rand();
		values2[j] = (float)(rand()) / rand();
	}

	for (int i = 0; i < nNAgent; i++) {

		ntrade[i] = nAgent[i] * nAgent[i] / 2;
		std::cout << "iteration " << i << " nAgent " << nAgent[i] << " ntrade " << ntrade[i] << std::endl;
		int numBlocks = ceil((ntrade[i] + blockSize - 1) / blockSize);

		MatrixGPU tempN(blockSize, 1, 0, 1);
		MatrixGPU tempN2(blockSize, 1, 0, 1);

		MatrixGPU CoresLinAgent(ntrade[i], 1);
		MatrixGPU CoresLinVoisin(ntrade[i], 1);
		MatrixGPU CoresMatLin(nAgent[i], nAgent[i], -1);
		MatrixGPU CoresLinTrans(ntrade[i], 1);


		int indice = 0;
		int idVoisin = 0;

		for (int idAgent = 0; idAgent < nAgent[i]; idAgent++) {
			int Nvoisinmax = nAgent[i] / 2;
			if (idAgent < Nvoisinmax) {
				idVoisin = 0;
			}
			else {
				idVoisin = nAgent[i] / 2;
			}
			for (int voisin = idVoisin; voisin < Nvoisinmax; voisin++) {
				CoresLinAgent.set(indice, 0, idAgent);
				CoresLinVoisin.set(indice, 0, voisin);
				CoresMatLin.set(idAgent, voisin, indice);
				indice = indice + 1;
			}
		}
		for (int lin = 0;lin < ntrade[i];lin++) {
			int i = CoresLinAgent.get(lin, 0);
			int j = CoresLinVoisin.get(lin, 0);
			int k = CoresMatLin.get(j, i);
			CoresLinTrans.set(lin, 0, k);
		}
		CoresLinAgent.transferGPU();
		CoresLinVoisin.transferGPU();
		CoresMatLin.transferGPU();
		CoresLinTrans.transferGPU();
		clock_t t = clock();

		for (int simu = 0; simu < nSimu; simu++) {
			MatrixGPU Tlocal_pre(ntrade[i], 1, values1[simu], 1);
			MatrixGPU Tlocal(ntrade[i], 1, values2[simu], 1);
			
			hipDeviceSynchronize();
			time = 0;
			for (int repet = 0; repet < nRepet; repet++) {
				MatrixGPU Tlocal_preCopy(Tlocal_pre);
				MatrixGPU TlocalCopy(Tlocal);
				MatrixGPU CoresLinTransCopy(CoresLinTrans);
				MatrixGPU tempNCopy(tempN);
				MatrixGPU tempN2Copy(tempN2);
				hipDeviceSynchronize();
				a = std::chrono::high_resolution_clock::now();
				float resS = TlocalCopy.max2(&Tlocal_preCopy);
				updateDiffGPU<< <numBlocks, blockSize >> > (tempNCopy._matrixGPU, TlocalCopy._matrixGPU, CoresLinTransCopy._matrixGPU, ntrade[i]);
				float resR = tempNCopy.max2();
				hipDeviceSynchronize();
				b = std::chrono::high_resolution_clock::now();
				time += std::chrono::duration_cast<std::chrono::nanoseconds>(b - a).count();
			}
			temps.set(i, simu, (float)time / nRepet);
		}
	}
	temps.saveCSV(fileName, mode);

}

void testADMMGPUConst1TimeCalcRes() {
	std::string fileName = "TempsResL8.csv";

	std::chrono::high_resolution_clock::time_point a;
	std::chrono::high_resolution_clock::time_point b;
	unsigned int time;
	std::ios_base::openmode mode = std::fstream::in | std::fstream::out | std::fstream::app;
	const int nNAgent = 7;
	const int nSimu = 100;
	const int nRepet = 10;
	int nAgent[nNAgent] = { 10, 100, 500, 1000, 5000, 10000, 40000 }; // autant conso que de prod, la derni�re veleur ne "marche" pas (trop rapide)
	int ntrade[nNAgent];
	int blockSize = 256;
	float values1[nSimu];
	float values2[nSimu];

	MatrixCPU temps(nNAgent, nSimu, 0);
	
	for (int j = 0; j < nSimu;j++) {
		values1[j] = (float)(rand()) / rand();
		values2[j] = (float)(rand()) / rand();
	}

	for (int i = 0; i < nNAgent; i++) {

		ntrade[i] = nAgent[i] * nAgent[i] / 2;
		std::cout << "iteration " << i << " nAgent " << nAgent[i] << " ntrade " << ntrade[i] << std::endl;
		int numBlocks = ceil((ntrade[i] + blockSize - 1) / blockSize);

		MatrixGPU tempN(blockSize, 1, 0, 1);
		MatrixGPU tempN2(blockSize, 1, 0, 1);

		clock_t t = clock();

		for (int simu = 0; simu < nSimu; simu++) {
			MatrixGPU Tlocal_pre(ntrade[i], 1, values1[simu], 1);
			MatrixGPU Tlocal(ntrade[i], 1, values2[simu], 1);
			MatrixGPU P(nAgent[i], 1, values1[simu], 1);
			MatrixGPU Tmoy(nAgent[i], 1, values2[simu], 1);

			hipDeviceSynchronize();
			time = 0;
			for (int repet = 0; repet < nRepet; repet++) {
				MatrixGPU Tlocal_preCopy(Tlocal_pre);
				MatrixGPU TlocalCopy(Tlocal);
				MatrixGPU PCopy(P);
				MatrixGPU TmoyCopy(Tmoy);
				MatrixGPU tempNCopy(tempN);
				MatrixGPU tempN2Copy(tempN2);
				hipDeviceSynchronize();
				a = std::chrono::high_resolution_clock::now();
				
				float d11 = Tlocal.max2(&Tlocal_pre);
				float d22 = P.max2(&Tmoy);
			
				hipDeviceSynchronize();
				b = std::chrono::high_resolution_clock::now();
				time += std::chrono::duration_cast<std::chrono::nanoseconds>(b - a).count();
			}


			/*hipEventCreate(&stop);
			hipEventRecord(stop, 0);
			hipEventSynchronize(stop);
			hipEventElapsedTime(&elapsedTime, start, stop);*/
			temps.set(i, simu, (float)time / nRepet);
		}
	}
	temps.saveCSV(fileName, mode);

}



