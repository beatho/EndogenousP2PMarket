#include "hip/hip_runtime.h"
#include "../head/StudyCaseDCGrid.h"
#include "../head/StudyCaseDCGrid.cuh"


void StudyCaseDCGrid::setGridFromFile(const std::string& path, MatrixCPU* fileCoresBus)
{
	MatrixCPU matFile(_nLine, 4);
	matFile.setFromFile(path);
	_nLineConstraint = 0;
	for (int i = 0; i < _nLine; i++) {
		int nodeFromFile = matFile.get(i, 0);
		int nodeToFile = matFile.get(i, 1);
		float react = matFile.get(i, 2); 
		if (react == 100000) { // cas pas de donn�e dans le r�seau europ�en
			react = 100;// que faire de ces "non" donn�es ?
		}
		float limit = matFile.get(i, 3) / _Sbase;

		int nodeFrom = fileCoresBus->get(nodeFromFile, 0);
		int nodeTo = fileCoresBus->get(nodeToFile, 0);

		//std::cout << " Ligne numero " << i << " entre bus " << nodeFromFile << " et " << nodeToFile << " dans le fichier mais en vrai c'est entre " << nodeFrom << " et " << nodeTo << " limite " << limit << " react " << react << std::endl;
		//std::cout << " Ligne numero " << i << " entre bus " << nodeFromFile << " et " << nodeToFile << " dans le fichier mais en vrai c'est entre " << nodeFrom << " et " << nodeTo << std::endl;
		
		_LineImpedance.set(i, i, react);
		_CoresBusLine.set(nodeFrom, i, 1);
		_CoresBusLine.set(nodeTo, i, -1);
		_CoresLineBus.set(i, 0, nodeFrom);
		_CoresLineBus.set(i, 1, nodeTo);
		if (limit > 0) {
			_nLineConstraint++;
			_lineLimits.set(i, 0, limit);
			_indiceLineConstraint.push_back(i);
		}
		else {
			_lineLimits.set(i, 0, LINELIMITMAX);
			_indiceLineNonConstraint.push_back(i);
		}
	}
}
void StudyCaseDCGrid::setBusFromFile(const std::string& path, MatrixCPU* fileCoresBus)
{
	int zone = 0;
	std::ifstream myfile(path, std::ios::in);
	bool found = false;
	int indice = zone;
	if (myfile)
	{
		for (int i = 0; i < _nBus; i++) {		
			int idAgent;
			int idBus;
			std::string country;
			myfile >> idAgent;
			myfile >> idBus;
			myfile >> country;
			fileCoresBus->set(idAgent, 0, idBus);
			
			found = false;
			indice = zone;
			for (int j = 0; j < _nameZone.size(); j++) {
				std::string value = _nameZone[j];
				if (!value.compare(country)) {
					found = true;
					indice = j;
					break;
				}
			}
			if (!found) {
				_nameZone.push_back(country);
				zone++;
			}
			_zoneBus.set(idAgent, 0, indice);

		}
		myfile.close();
	}
	else {
		throw std::invalid_argument("can't open this file");
	}
}
void StudyCaseDCGrid::CalcGridSensi()
{
	
	MatrixCPU temp1(_nLine, _nBus); // BC^T 
	MatrixCPU temp2(_nBus, _nBus); // CBC^T
	MatrixCPU temp3(_nBus, _nBus); // (CBC^T)^-1 avec mise � 0 de la ligne et colonne du noued de ref
	MatrixCPU temp33(_nBus - 1, _nBus - 1); //(CBC ^ T) ^ -1 sans la ligne et colonne du noued de ref
	MatrixCPU temp22(_nBus - 1, _nBus - 1); // on enl�ve la ligne du noeud de ref�rence
	MatrixCPU result(_nBus - 1, _nBus - 1);
	
	//MatrixCPU identity(_nBus - 1, _nBus - 1);
	//identity.setEyes(1);
	//std::cout << "nBus " << _nBus << " nLine " << _nLine <<  std::endl;



	temp1.multiplyTrans(&_LineImpedance, &_CoresBusLine);
	temp2.multiply(&_CoresBusLine, &temp1);
	
	temp2.getBloc(&temp22, 1, _nBus, 1, _nBus);

	//temp22.display();
	

	if (_invertMethod==1) {
		
		MatrixGPU temp33GPU(_nBus-1, _nBus-1, 0, 1);
		
		MatrixGPU temp22GPU(temp22,1);	
		temp33GPU.invertGaussJordan(&temp22GPU);
		temp33GPU.toMatCPU(temp33);


		//result.multiply(&temp33,&temp22);
		//float err = result.distance2(&identity);

		//std::cout << "err GPU " << err << std::endl;


	}
	else if(_invertMethod==2)
	{
		temp33.invertGaussJordan(&temp22);
		//result.multiply(&temp33, &temp22);
		//float err = result.distance2(&identity);

		//std::cout << "err CPU " << err << std::endl;
	}
	else {
	
		temp33.invertGaussJordan(&temp22);
		//result.multiply(&temp33, &temp22);
		//float err = result.distance2(&identity);

		//std::cout << "err Eigen " << err << std::endl;
	}

	temp3.setBloc(1, _nBus, 1, _nBus, &temp33);
	_SensiBusLine.multiply(&temp1, &temp3);

	

}
void StudyCaseDCGrid::ReduceSensi()
{
	
	_lineLimitsReduce = MatrixCPU(_nLineConstraint, 1);
	_SensiBusLineReduce = MatrixCPU(_nLineConstraint, _nBus); // Ared
	_CoresLineBusReduce = MatrixCPU(_nLineConstraint, 2);
	_indiceLineConstraint.clear();
	_indiceLineNonConstraint.clear();
	int indice = 0;
	for (int i = 0; i < _nLine; i++) {
		float lim = _lineLimits.get(i, 0);
		if (lim != 0 && lim != LINELIMITMAX) {
			_indiceLineConstraint.push_back(i);
			_lineLimitsReduce.set(indice, 0, lim);
			_CoresLineBusReduce.set(indice, 0, _CoresLineBus.get(i, 0));
			_CoresLineBusReduce.set(indice, 1, _CoresLineBus.get(i, 1));
			for (int j = 0; j < _nBus; j++) {
				_SensiBusLineReduce.set(indice, j, _SensiBusLine.get(i, j));
			}
			indice++;
		}
		else {
			_indiceLineNonConstraint.push_back(i);
			_lineLimits.set(i, 0, LINELIMITMAX);
		}
		
	}
}

float StudyCaseDCGrid::rand1() const
{
	float a = (float)(rand()) / ((float)(RAND_MAX));
	return a;
}
int StudyCaseDCGrid::randab(int a, int b) const
{
	return a + (rand() % (b - a));
}
int StudyCaseDCGrid::getNFileline(std::string nameFile)
{
	int number_of_lines = 0;
	std::string line;
	std::ifstream myfile(nameFile);

	while (std::getline(myfile, line))
		++number_of_lines;
	return number_of_lines;
}


StudyCaseDCGrid::StudyCaseDCGrid()
{
	
	_nBus = 2;
	_nLine = 1;
	_nLineConstraint = 0;
	float x = -0.01;
	
	_LineImpedance = MatrixCPU(_nLine, _nLine, -1 / x);
	_CoresLineBus = MatrixCPU(_nLine, 2);
	_CoresLineBus.set(0, 0, 0);
	_CoresLineBus.set(0, 1, 1);
	_CoresBusLine = MatrixCPU(_nBus, _nLine);
	_CoresBusLine.set(0, 0, 1);
	_CoresBusLine.set(1, 0, -1);
	_SensiBusLine = MatrixCPU(_nLine, _nBus); 
	_lineLimits = MatrixCPU(_nLine, 1, 0);
	
	//std::cout << "gridSensi " << std::endl;
	CalcGridSensi();
	//std::cout << "reduce " << std::endl;
	ReduceSensi();
	 
}
StudyCaseDCGrid::StudyCaseDCGrid(const StudyCaseDCGrid& s)
{
	clock_t t = clock();

	_nLine = s._nLine;
	_nBus = s._nBus;
	_nLineConstraint = s._nLineConstraint;
	_name = s._name;
	toReduce = s.toReduce;
	_Zbase = s._Zbase;
	_Sbase = s._Sbase;

	_lineLimitsReduce = s._lineLimitsReduce;
	_SensiBusLine = s._SensiBusLine;
	_SensiBusLineReduce = s._SensiBusLineReduce;



	_LineImpedance = s._LineImpedance; // B
	_CoresBusLine = s._CoresBusLine; // C
	_lineLimits = s._lineLimits; // l

	// min
	_lineLimitsChange = s._lineLimitsChange;
	lineMin = s.lineMin;
	lineoffset = s.lineoffset;

	_CoresLineBus = s._CoresLineBus;



	t = clock() - t;
	_timeInit = (float)t / CLOCKS_PER_SEC;
}
StudyCaseDCGrid& StudyCaseDCGrid::operator= (const StudyCaseDCGrid& s)
{
	clock_t t = clock();
	//std::cout << " Copie egalite  DC " << std::endl;
	_nLine = s._nLine;
	_nBus = s._nBus;
	_nLineConstraint = s._nLineConstraint;
	_name = s._name;
	toReduce = s.toReduce;
	_Zbase = s._Zbase;
	_Sbase = s._Sbase;

	_lineLimitsReduce = s._lineLimitsReduce;
	_SensiBusLine = s._SensiBusLine;
	_SensiBusLineReduce = s._SensiBusLineReduce;



	_LineImpedance = s._LineImpedance; // B
	_CoresBusLine = s._CoresBusLine; // C
	_lineLimits = s._lineLimits; // l

	// min
	_lineLimitsChange = s._lineLimitsChange;
	lineMin = s.lineMin;
	lineoffset = s.lineoffset;

	_CoresLineBus = s._CoresLineBus;



	t = clock() - t;
	_timeInit = (float)t / CLOCKS_PER_SEC;

	return *this;
}


void StudyCaseDCGrid::genGridFromFile(std::string path, bool alreadyDefine)
{
	// grid 
	_nBus = 1494;
	_nLine = 2156;
	//std::ios_base::openmode mode = std::fstream::in | std::fstream::out | std::fstream::app;
	std::string fileName = path + "SensiBusLineEurope.txt";
	std::string fileName2 = path + "lineLimitEurope.txt";
	std::string fileName3 = path + "SensiBusLineReduceEurope.txt";
	std::string fileName4 = path + "lineLimitReduceEurope.txt";
	std::string pathGrid = path + "Network.txt";
	std::string pathBus = path + "BusAgent.txt"; // corespondance entre les "idBus" du fichier et celui du code (exemple commence � 0 ou � 1)

	
	_LineImpedance = MatrixCPU(_nLine, _nLine); // B
	_CoresBusLine = MatrixCPU(_nBus, _nLine); // C
	_CoresLineBus = MatrixCPU(_nLine, 2); // from, to
	_lineLimits = MatrixCPU(_nLine, 1);
	_zoneBus = MatrixCPU(_nBus, 1);
	_SensiBusLine = MatrixCPU(_nLine, _nBus);
	MatrixCPU fileCoresBus(_nBus, 1);

	if (alreadyDefine) {
		setBusFromFile(pathBus, &fileCoresBus);
		int idBusMax = fileCoresBus.max2();
		MatrixCPU fileBusAgent(idBusMax + 1, 1, -1); // si reste � -1, le bus n'existe pas
		for (int i = 0; i < _nBus; i++) {
			int bus = fileCoresBus.get(i, 0);
			fileBusAgent.set(bus, 0, i);
		}
		setGridFromFile(pathGrid, &fileBusAgent);


		_nLineConstraint = getNFileline(fileName4);
		_SensiBusLineReduce = MatrixCPU(_nLineConstraint, _nBus);
		_lineLimitsReduce = MatrixCPU(_nLineConstraint, 1);
		_SensiBusLine.setFromFile(fileName);
		_lineLimits.setFromFile(fileName2);
		_SensiBusLineReduce.setFromFile(fileName3);
		_lineLimitsReduce.setFromFile(fileName4);
	}
	else {

		std::ios_base::openmode mode = std::fstream::in | std::fstream::out | std::fstream::app;
		std::string pathGrid = path + "Network.txt";
		std::string pathBus = path + "BusAgent.txt"; // corespondance entre les "idBus" du fichier et celui du code (exemple commence � 0 ou � 1)


		setBusFromFile(pathBus, &fileCoresBus);

		int idBusMax = fileCoresBus.max2();
		MatrixCPU fileBusAgent(idBusMax + 1, 1, -1); // si reste � -1, le bus n'existe pas


		for (int i = 0; i < _nBus; i++) {
			int bus = fileCoresBus.get(i, 0);
			fileBusAgent.set(bus, 0, i);
		}


		setGridFromFile(pathGrid, &fileBusAgent);



		CalcGridSensi();
		ReduceSensi();
		_SensiBusLine.saveCSV(fileName, mode);
		_lineLimits.saveCSV(fileName2, mode);
		_SensiBusLineReduce.saveCSV(fileName3, mode);
		_lineLimitsReduce.saveCSV(fileName4, mode);
	}
}
void StudyCaseDCGrid::genLineLimit(int nLine, float limit, float dlLimit)
{
	//std::cout << "genLineLimit" << std::endl;
	if (nLine > _nLine) {
		throw std::invalid_argument("nLine is too big");
	}
	if (_nLine < 0) {
		throw std::invalid_argument("nLine must be positive");
	}
	if (_indiceLineNonConstraint.size() + _indiceLineConstraint.size() != _nLine) {
		_indiceLineNonConstraint.clear();
		_indiceLineConstraint.clear();
		//std::cout << _indiceLineNonConstraint.size() << " " << _indiceLineConstraint.size() << " " << _nLine << std::endl;
		//_lineLimits.display();
		for (int i = 0; i < _nLine; i++) {
			float lim = _lineLimits.get(i, 0);
			if (lim != 0 && lim != LINELIMITMAX) {
				_indiceLineConstraint.push_back(i);
			}
			else {
				_indiceLineNonConstraint.push_back(i);
			}

		}
	}
	
	if (nLine > _nLineConstraint) // doit augmenter le nombre de ligne contrainte
	{
		int dl = nLine - _nLineConstraint;
		for (int i = 0; i < dl; i++) {
			int indice = rand() % (_indiceLineNonConstraint.size());
			int j = _indiceLineNonConstraint[indice];
			_indiceLineNonConstraint.erase(_indiceLineNonConstraint.begin() + indice);
			_indiceLineConstraint.push_back(j);
			float l = limit + 2 * dlLimit * (rand1() - 0.5);
			_lineLimits.set(j, 0, l);
		}

	}
	else { // doit diminuer le nombre de ligne contrainte
		if (nLine == 0) {
			int dl = _nLineConstraint;
			for (int i = 0; i < dl; i++) {
				int j = _indiceLineConstraint[dl - i - 1];
				_indiceLineConstraint.pop_back();
				_indiceLineNonConstraint.push_back(j);
				_lineLimits.set(j, 0, 0);
			}
		}
		else {
			int dl = _nLineConstraint - nLine;
			for (int i = 0; i < dl; i++) {
				//std::cout << "-";
				int indice = rand() % _indiceLineConstraint.size();
				int j = _indiceLineConstraint[indice];

				_indiceLineConstraint.erase(_indiceLineConstraint.begin() + indice);
				_indiceLineNonConstraint.push_back(j);
				_lineLimits.set(j, 0, 0);
			}
		}

	}
	//std::cout << std::endl;
	_nLineConstraint = _indiceLineConstraint.size();
	ReduceSensi();

}



void StudyCaseDCGrid::Set39Bus(std::string path, bool alreadyDefine)
{
	clock_t t = clock();

	// grid 
	_nBus = 39;
	_nLine = 46;
	_Sbase = 1; //MW
	float zoneBus[39] = { 0, 0, 0, 2, 2, 2, 2, 2, 2, 2,
						  2, 2, 2, 2, 3, 3, 3, 0, 3, 3,
						  3, 3, 3, 3, 0, 1, 1, 1, 1, 0,
						  2, 2, 3, 3, 3, 3, 0, 1, 0 };
	
	std::string filename = path + "Network39.txt";
	_LineImpedance = MatrixCPU(_nLine, _nLine); // B
	_CoresBusLine = MatrixCPU(_nBus, _nLine); // C
	_CoresLineBus = MatrixCPU(_nLine, 2); // from, to
	_lineLimits = MatrixCPU(_nLine, 1);
	_SensiBusLine = MatrixCPU(_nLine, _nBus);
	
	_zoneBus = MatrixCPU(_nBus, 1);
	std::ios_base::openmode mode = std::fstream::in | std::fstream::out | std::fstream::app;

	MatrixCPU fileCoresBus(_nBus + 1, 1);
	for (int i = 0; i < _nBus + 1; i++) {
		fileCoresBus.set(i, 0, i - 1);
	}
	setGridFromFile(filename, &fileCoresBus);
	for (int id = 0; id < _nBus; id++) {
			_zoneBus.set(id, 0, zoneBus[id]);
	}

	std::string fileName = path + "SensiBusLine39node.txt";
	std::string fileName3 = path + "SensiBusLineReduce39node.txt";
	std::string fileName4 = path + "lineLimitReduce39node.txt";

	if (alreadyDefine) {
		_SensiBusLineReduce = MatrixCPU(_nLineConstraint, _nBus);
		_lineLimitsReduce = MatrixCPU(_nLineConstraint, 1);
		_SensiBusLine.setFromFile(fileName);
		_SensiBusLineReduce.setFromFile(fileName3);
		_lineLimitsReduce.setFromFile(fileName4);
	}
	else {

		CalcGridSensi();
		//_SensiPower.multiply(&_SensiBusLine, &_CoresBusAgent);
		ReduceSensi();
		_SensiBusLine.saveCSV(fileName, mode, 0, " ");
		_SensiBusLineReduce.saveCSV(fileName3, mode, 0, " ");
		_lineLimitsReduce.saveCSV(fileName4, mode, 0, " ");
	}
	t = clock() - t;
	_timeInit = (float)t / CLOCKS_PER_SEC;
}
void StudyCaseDCGrid::Set3Bus(std::string path) {
	clock_t t = clock();
	// grid 
	_nBus = 3;
	_nLine = 3;
	std::string filename = path + "Network3.txt";
	
	_LineImpedance = MatrixCPU(_nLine, _nLine); // B
	_CoresBusLine = MatrixCPU(_nBus, _nLine); // C
	_CoresLineBus = MatrixCPU(_nLine, 2); // from, to
	_lineLimits = MatrixCPU(_nLine, 1);
	_SensiBusLine = MatrixCPU(_nLine, _nBus); // A
	//std::ios_base::openmode mode = std::fstream::in | std::fstream::out | std::fstream::app;
	
	MatrixCPU fileCoresBus(_nBus, 1);
	for (int i = 0; i < _nBus; i++) {
		fileCoresBus.set(i, 0, i);
	}
	setGridFromFile(filename, &fileCoresBus);
	CalcGridSensi();

	/*for (int id = 0; id < _nAgent; id++) {
		_CoresBusAgent.set(fileCoresBus.get(BusAgent[id], 0), id, 1);
	}
	
	_SensiPower.multiply(&_SensiBusLine, &_CoresBusAgent);*/

	ReduceSensi();
	
	
	t = clock() - t;
	_timeInit = (float)t / CLOCKS_PER_SEC;
}
void StudyCaseDCGrid::Set4nodeBis(std::string path)
{
	// cas d'�tude pour simuler le cas d'EVA pendant son stage
	clock_t t = clock();
	
	// grid 
	_nBus = 4;
	_nLine = 4;
	std::string filename = path + "Network4.txt";
	
	_LineImpedance = MatrixCPU(_nLine, _nLine); // B
	_CoresBusLine = MatrixCPU(_nBus, _nLine); // C
	_CoresLineBus = MatrixCPU(_nLine, 2); // from, to
	_lineLimits = MatrixCPU(_nLine, 1);
	//_CoresBusAgent = MatrixCPU(_nBus, _nAgent);
	_SensiBusLine = MatrixCPU(_nLine, _nBus); // A
	//std::ios_base::openmode mode = std::fstream::in | std::fstream::out | std::fstream::app;



	MatrixCPU fileCoresBus(_nBus, 1);
	for (int i = 0; i < _nBus; i++) {
		fileCoresBus.set(i, 0, i);
	}
	setGridFromFile(filename, &fileCoresBus);


	/*for (int id = 0; id < _nAgent; id++) {
		_CoresBusAgent.set(fileCoresBus.get(BusAgent[id], 0), id, 1);
	}
	_SensiPower.multiply(&_SensiBusLine, &_CoresBusAgent);*/
	CalcGridSensi();
	
	ReduceSensi();

	

	
	t = clock() - t;
	_timeInit = (float)t / CLOCKS_PER_SEC;


}
void StudyCaseDCGrid::Set2nodeConstraint(float lim)
{
	clock_t t = clock();
	
	// grid 
	_nBus = 2;
	_nLine = 1;
	

	_LineImpedance = MatrixCPU(_nLine, _nLine); // B
	_CoresBusLine = MatrixCPU(_nBus, _nLine); // C
	_lineLimits = MatrixCPU(_nLine, 1);
	_SensiBusLine = MatrixCPU(_nLine, _nBus); // A
	_zoneBus = MatrixCPU(_nBus, 1);
	_CoresLineBus = MatrixCPU(_nLine, 2); // from, to
	

	/*_CoresBusAgent.set(0, 0, 1);
	_CoresBusAgent.set(1, 1, 1);*/
	
	
	_LineImpedance.set(0, 0, 1); //bii = 1; se simplifie
	

	_SensiBusLine.set(0, 1, -1);
	_lineLimits.set(0, 0, lim);

	
	_nLineConstraint = _nLine;
	_SensiBusLineReduce = MatrixCPU(_SensiBusLine);
	_lineLimitsReduce = MatrixCPU(_lineLimits);
	
	t = clock() - t;
	_timeInit = (float)t / CLOCKS_PER_SEC;
}

void StudyCaseDCGrid::SetEuropeP0(const std::string& path, bool alreadyDefine)
{
	clock_t t = clock();
	_nBus = 1494;
	_nLine = 2156;
	
	_LineImpedance = MatrixCPU(_nLine, _nLine); // B
	_CoresBusLine = MatrixCPU(_nBus, _nLine); // C
	_CoresLineBus = MatrixCPU(_nLine, 2); // from, to
	_lineLimits = MatrixCPU(_nLine, 1);
	_SensiBusLine = MatrixCPU(_nLine, _nBus); // A
	_zoneBus = MatrixCPU(_nBus, 1);
	
	std::ios_base::openmode mode = std::fstream::in | std::fstream::out | std::fstream::app;
	std::string fileName = path  + "SensiBusLineEurope.txt";
	std::string fileName2 = path + "lineLimitEurope.txt";
	std::string fileName3 = path + "SensiBusLineReduceEurope.txt";
	std::string fileName4 = path + "lineLimitReduceEurope.txt";

	if (alreadyDefine) {
		_nLineConstraint = getNFileline(fileName4);
		_SensiBusLineReduce = MatrixCPU(_nLineConstraint, _nBus);
		_lineLimitsReduce = MatrixCPU(_nLineConstraint, 1);
		_SensiBusLine.setFromFile(fileName);
		_lineLimits.setFromFile(fileName2);
		_SensiBusLineReduce.setFromFile(fileName3);
		_lineLimitsReduce.setFromFile(fileName4);
	}
	else {

		std::string pathGrid = path + "Network.txt";
		fileCoresBus = MatrixCPU(_nBus, 1);
		std::string pathBus = path + "BusAgent.txt"; // corespondance entre les "idBus" du fichier et celui du code (exemple commence � 0 ou � 1)
		setBusFromFile(pathBus, &fileCoresBus);
		int idBusMax = fileCoresBus.max2();
		MatrixCPU fileBusAgent(idBusMax + 1, 1, -1); // si reste � -1, le bus n'existe pas
		//std::cout << idBusMax << std::endl;
		for (int i = 0; i < _nBus; i++) {
			int bus = fileCoresBus.get(i, 0);
			fileBusAgent.set(bus, 0, i);
		}
		setGridFromFile(pathGrid, &fileBusAgent);
		CalcGridSensi();
		ReduceSensi();
		

		_SensiBusLine.saveCSV(fileName, mode);
		_lineLimits.saveCSV(fileName2, mode);
		_SensiBusLineReduce.saveCSV(fileName3, mode);
		_lineLimitsReduce.saveCSV(fileName4, mode);
	}
	//_SensiPower.display();
	t = clock() - t;
	//_timeInit = (float)t / CLOCKS_PER_SEC;
}
void StudyCaseDCGrid::SetStudyCaseDCGrid(std::string path, std::string name, int nBus, bool alreadyDefine)
{
	
	clock_t t = clock();

	std::string fileName = path + "SensiBusLine" + name + ".txt";
	std::string fileName2 = path + "lineLimit" + name + ".txt";
	std::string fileName3 = path + "SensiBusLineReduce" + name + ".txt";
	std::string fileName4 = path + "lineLimitReduce" + name + ".txt";
	
	_name = name;
	
	
	// grid 
	_nBus = nBus; //_nBus = _nCons;
	
	std::string pathGrid = path + "Network" + name + ".txt";
	_nLine = getNFileline(pathGrid);
	//std::cout << "nb de ligne " <<  _nLine << std::endl;
	_LineImpedance = MatrixCPU(_nLine, _nLine); // B
	_CoresBusLine = MatrixCPU(_nBus, _nLine); // C
	_lineLimits = MatrixCPU(_nLine, 1);
	_SensiBusLine = MatrixCPU(_nLine, _nBus); // A
	_zoneBus = MatrixCPU(_nBus, 1);
	_CoresLineBus = MatrixCPU(_nLine, 2);
	
	std::ios_base::openmode mode = std::fstream::in | std::fstream::out | std::fstream::app;
	

	if (alreadyDefine) {
		_nLineConstraint = getNFileline(fileName4);
		_SensiBusLineReduce = MatrixCPU(_nLineConstraint, _nBus);
		_lineLimitsReduce = MatrixCPU(_nLineConstraint, 1);
		_SensiBusLine.setFromFile(fileName);
		_lineLimits.setFromFile(fileName2);
		_SensiBusLineReduce.setFromFile(fileName3);
		_lineLimitsReduce.setFromFile(fileName4);
	}
	else {

		std::string pathGrid = path + "Network" + name + ".txt";
		fileCoresBus = MatrixCPU(_nBus, 1);
		std::string pathBus = path + "BusAgent" + name + ".txt"; // corespondance entre les "idBus" du fichier et celui du code (exemple commence � 0 ou � 1)
		
		setBusFromFile(pathBus, &fileCoresBus);
		int idBusMax = fileCoresBus.max2();
		MatrixCPU fileBusAgent(idBusMax + 1, 1, -1); // si reste � -1, le bus n'existe pas
		//std::cout << idBusMax << std::endl;
		
		for (int i = 0; i < _nBus; i++) {
			int bus = fileCoresBus.get(i, 0);
			fileBusAgent.set(bus, 0, i);
		}
		
		setGridFromFile(pathGrid, &fileBusAgent);
	
		CalcGridSensi();

		ReduceSensi();

		/*std::cout << _nLineConstraint << std::endl;
		for (int i = 0; i < _nAgent; i++) {
			if (i < _nCons) { // le bus correspond directement pour les conso
				int bus = i;
				_CoresBusAgent.set(bus, i, 1);
			}
			else {
				int idGen = i - _nCons;
				int bus = fileBusAgent.get(GenBus.get(idGen, 0), 0);
				_CoresBusAgent.set(bus, i, 1);
			}
		}
		_SensiPower.multiply(&_SensiBusLine, &_CoresBusAgent);
		*/
		
		_SensiBusLine.saveCSV(fileName, mode);
		_lineLimits.saveCSV(fileName2, mode);
		_SensiBusLineReduce.saveCSV(fileName3, mode);
		_lineLimitsReduce.saveCSV(fileName4, mode);
	}
	//_SensiPower.display();
	t = clock() - t;
	//_timeInit = (float)t / CLOCKS_PER_SEC;
}

void StudyCaseDCGrid::setFromInterface(StudyCaseInterface* interface){
	clock_t t = clock();

	_name = interface->getName();
	
	MatrixCPU infoCase = interface->getInfoCase();
	_Sbase = infoCase.get(0, Sbase_ind);

	// grid 
	_nBus = interface->getB(); //_nBus = _nCons;
	_nLine = interface->getL();
	//std::cout << "nb de ligne " <<  _nLine << std::endl;
	_LineImpedance = MatrixCPU(_nLine, _nLine); // B
	_CoresBusLine = MatrixCPU(_nBus, _nLine); // C
	_lineLimits = MatrixCPU(_nLine, 1);
	_SensiBusLine = MatrixCPU(_nLine, _nBus); // A
	_zoneBus = MatrixCPU(_nBus, 1);
	_CoresLineBus = MatrixCPU(_nLine, 2);
	
	//std::ios_base::openmode mode = std::fstream::in | std::fstream::out | std::fstream::app;
	
	MatrixCPU branchCase = interface->getBranchCase();
	MatrixCPU busCase    = interface->getBusCase();

	_nLineConstraint = 0;
	for (int i = 0; i < _nLine; i++) {
		int nodeFrom = branchCase.get(i, From_ind);
		int nodeTo   = branchCase.get(i, To_ind);
		float react = branchCase.get(i, ZsIm_ind); 
		
		float limit = branchCase.get(i, lim_ind) / _Sbase;

		_LineImpedance.set(i, i, -react);
		_CoresBusLine.set(nodeFrom, i, 1);
		_CoresBusLine.set(nodeTo, i, -1);
		_CoresLineBus.set(i, 0, nodeFrom);
		_CoresLineBus.set(i, 1, nodeTo);
		if (limit > 0) {
			_nLineConstraint++;
			_lineLimits.set(i, 0, limit);
			_indiceLineConstraint.push_back(i);
		}
		else {
			_lineLimits.set(i, 0, LINELIMITMAX);
			_indiceLineNonConstraint.push_back(i);
		}
	}

	CalcGridSensi();

	ReduceSensi();

	/*std::cout << _nLineConstraint << std::endl;
	for (int i = 0; i < _nAgent; i++) {
		if (i < _nCons) { // le bus correspond directement pour les conso
			int bus = i;
			_CoresBusAgent.set(bus, i, 1);
		}
		else {
			int idGen = i - _nCons;
			int bus = fileBusAgent.get(GenBus.get(idGen, 0), 0);
			_CoresBusAgent.set(bus, i, 1);
		}
	}
	_SensiPower.multiply(&_SensiBusLine, &_CoresBusAgent);
	*/
		
	
	//_SensiPower.display();
	t = clock() - t;
	//_timeInit = (float)t / CLOCKS_PER_SEC;
	
}




void StudyCaseDCGrid::setLineLimitMin(float min)
{
	if (lineoffset) {
		lineoffset = 0;
	}
	_lineLimitsChange = getLineLimit();
	lineMin = min;
	for (int l = 0; l < getNLine(); l++) {
		if (_lineLimitsChange.get(l, 0) < lineMin) {
			_lineLimitsChange.set(l, 0, lineMin);
		}
	}
}
void StudyCaseDCGrid::setLineLimitRelaxation(float eps)
{
	if (lineMin) {
		lineMin = 0;
	}
	_lineLimitsChange = getLineLimit();
	lineoffset = eps;
	for (int l = 0; l < getNLine(); l++) {
		
		_lineLimitsChange.increment(l, 0, -eps);
		
	}
}
void StudyCaseDCGrid::setLineLimit(int line, float limit)
{
	if (line > _nLine) {
		throw std::invalid_argument("this line doesn't exist");
	}
	else {
		float oldLimit = _lineLimits.get(line, 0);
		if (oldLimit == LINELIMITMAX) {
			_nLineConstraint++;
		}
		_lineLimits.set(line, 0, limit/_Sbase);
		ReduceSensi();
	}
}


MatrixCPU StudyCaseDCGrid::getPowerSensiBus(bool force) const
{
	if (force) {
		return _SensiBusLine;
	}
	if (toReduce) {
		return _SensiBusLineReduce;
	}
	else {
		return _SensiBusLine;
	}
	
}
MatrixCPU StudyCaseDCGrid::getPowerSensiBusReduce() const
{
	return _SensiBusLineReduce;
}
MatrixCPU StudyCaseDCGrid::getLineLimit() const
{
	if (lineMin || lineoffset) {
		return _lineLimitsChange;
	} 
	else if (toReduce) {
		return _lineLimitsReduce;
	}
	else {
		return _lineLimits;
	}
	
}
MatrixCPU StudyCaseDCGrid::getCoresLineBus(bool force) const
{
	if (toReduce && !force) {
		return _CoresLineBusReduce;
	}
	else {
		return _CoresLineBus;
	}
}
MatrixCPU StudyCaseDCGrid::getfileCoresBus() const
{
	return fileCoresBus;
}
MatrixCPU StudyCaseDCGrid::getZones() const
{
	return _zoneBus;
}
float StudyCaseDCGrid::getTimeInit() const
{
	return _timeInit;
}
int StudyCaseDCGrid::getNLine(bool force) const
{
	if (force) {
		return _nLine;
	}
	if (toReduce) {
		return _nLineConstraint;
	}
	else {
		return _nLine;
	}
	
}
int StudyCaseDCGrid::getNLineConstraint() const
{
	return _nLineConstraint;
}
int StudyCaseDCGrid::getNBus() const
{
	return _nBus;
}
std::string StudyCaseDCGrid::getName() const
{
	return _name;
}



void StudyCaseDCGrid::saveCSV(const std::string& fileName)
{

	std::ios_base::openmode mode = std::fstream::in | std::fstream::out | std::fstream::app;
	MatrixCPU nombre(1, 3);
	nombre.set(0, 0, _nLine);
	nombre.set(0, 1, _nLineConstraint);
	nombre.set(0, 2, _nBus);
	nombre.saveCSV(fileName, mode);


	MatrixCPU temp2(1, _nLineConstraint);
	temp2.addTrans(&_lineLimitsReduce);
	temp2.saveCSV(fileName, mode);
	_SensiBusLineReduce.saveCSV(fileName, mode);

}

void StudyCaseDCGrid::display(int type) const
{
	std::cout << "Study Case : " << _nBus << " bus and " << _nLine << " lines " << std::endl;
	std::cout << "and " << _nLineConstraint << " and reduced " << toReduce << std::endl;

	if (_nLine < 100 && _nBus < 100) {
		std::cout << " B :" << std::endl;
		_LineImpedance.display();
		std::cout << " C :" << std::endl;
		_CoresBusLine.display();
		if (toReduce) {
			std::cout << " Line limit :" << std::endl;
			_lineLimitsReduce.display();
			std::cout << " Sensibility :" << std::endl;
			_SensiBusLineReduce.display();
		}
		else {
			std::cout << " Line limit :" << std::endl;
			_lineLimits.display();
			std::cout << " Sensibility :" << std::endl;
			_SensiBusLine.display();
		}
		
	}
	
	
	if(type ==1){

		MatrixCPU temp1(_nLine, _nBus); // BC^T 
		MatrixCPU temp2(_nBus, _nBus); // CBC^T
		MatrixCPU temp33(_nBus - 1, _nBus - 1); //(CBC ^ T) ^ -1 sans la ligne et colonne du noued de ref
		MatrixCPU temp22(_nBus - 1, _nBus - 1); // on enl�ve la ligne du noeud de ref�rence

		MatrixCPU LineImpedance(_LineImpedance);
		MatrixCPU CoresBusLine(_CoresBusLine);

		temp1.multiplyTrans(&LineImpedance, &CoresBusLine);
		temp2.multiply(&CoresBusLine, &temp1);
		temp2.getBloc(&temp22, 1, _nBus, 1, _nBus);
		temp33.invertGaussJordan(&temp22);
		std::cout << " BC^T :" << std::endl;
		temp1.display();
		std::cout << " CBC^T :" << std::endl;
		temp2.display();
		std::cout << " (CBC^T)^-1 :" << std::endl;
		temp33.display();
	
	}
	
}

void StudyCaseDCGrid::displayLineCores(MatrixCPU* g, bool all)
{
	std::cout << "Line correspendance : " << std::endl;
	
	MatrixCPU Cores(getCoresLineBus());
	//Cores.display();
	
	MatrixCPU Limit(getLineLimit());
	//Limit.display();
	if (all) {
		if (Cores.getNLin() == 0) {
			for (int l = 0; l < getNLine(); l++) {
				if (fabs(g->get(l, 0)) > Limit.get(l, 0)) {
					std::cout << "******* OverFlow : Line n " << l << " line limit " << Limit.get(l, 0)
						<< " flow " << g->get(l, 0) << "********" << std::endl;
				}
				else if (Limit.get(l, 0) - fabs(g->get(l, 0)) < 0.1) {
					std::cout << "+++ Close bounds : Line n " << l << " line limit " << Limit.get(l, 0)
						<< " flow " << g->get(l, 0) << "+++" << std::endl;
				}
				else {
					std::cout << "Line n " << l << " line limit " << Limit.get(l, 0)
						<< " flow " << g->get(l, 0) << std::endl;
				}
				
			}
		}
		else {
			for (int l = 0; l < getNLine(); l++) {
				if (fabs(g->get(l, 0)) > Limit.get(l, 0)) {
					std::cout << "********* OverFlow :Line n " << l << " from node " << Cores.get(l, 0)
						<< " to node " << Cores.get(l, 1) << " line limit " << Limit.get(l, 0)
						<< " flow " << g->get(l, 0) << "********" << std::endl;
				}
				else if (Limit.get(l, 0) - fabs(g->get(l, 0)) < 0.1) {
					std::cout << "+++ Close bounds :Line n " << l << " from node " << Cores.get(l, 0)
						<< " to node " << Cores.get(l, 1) << " line limit " << Limit.get(l, 0)
						<< " flow " << g->get(l, 0) << std::endl;
				}
				else {
					std::cout << "Line n " << l << " from node " << Cores.get(l, 0)
						<< " to node " << Cores.get(l, 1) << " line limit " << Limit.get(l, 0)
						<< " flow " << g->get(l, 0) << std::endl;
				}
				
			}
		}
	}
	else {
		if (Cores.getNLin() == 0) {
			for (int l = 0; l < getNLine(); l++) {
				if (fabs(g->get(l, 0)) > Limit.get(l, 0)) {
					std::cout << "********* OverFlow : Line n " << l << " line limit " << Limit.get(l, 0)
						<< " flow " << g->get(l, 0) << "*********"<< std::endl;
				}
				else if (Limit.get(l, 0) - fabs(g->get(l, 0)) < 0.1) {
					std::cout << "+++ Close bounds  : Line n " << l << " line limit " << Limit.get(l, 0)
						<< " flow " << g->get(l, 0) << "+++" << std::endl;
				}
				
			}
		}
		else {
			for (int l = 0; l < getNLine(); l++) {
				if (fabs(g->get(l, 0)) > Limit.get(l, 0)) {
					std::cout << "********* OverFlow :Line n " << l << " from node " << Cores.get(l, 0)
						<< " to node " << Cores.get(l, 1) << " line limit " << Limit.get(l, 0)
						<< " flow " << g->get(l, 0) << "*********" << std::endl;
				}
				else if (Limit.get(l, 0) - fabs(g->get(l, 0)) < 0.1) {
					std::cout << "+++ Close bounds :Line n " << l << " from node " << Cores.get(l, 0)
						<< " to node " << Cores.get(l, 1) << " line limit " << Limit.get(l, 0)
						<< " flow " << g->get(l, 0) << std::endl;
				}
				
			}
		}
	}
	
	
}

StudyCaseDCGrid::~StudyCaseDCGrid()
{
#ifdef DEBUG_DESTRUCTOR
	std::cout << "case destructor" << std::endl;
#endif

}


