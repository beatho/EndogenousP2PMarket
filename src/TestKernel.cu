#include "hip/hip_runtime.h"
﻿
#include "../head/TestKernel.cuh"


// --------------------------------- Q part -------------------------------------------------------

float testCalculQpart(int method) {

	std::string fileName = "TempsQpart"+std::to_string(method) + ".csv";
	//float elapsedTime;
	std::chrono::high_resolution_clock::time_point a;
	std::chrono::high_resolution_clock::time_point b;
	float time;
	std::ios_base::openmode mode = std::fstream::in | std::fstream::out | std::fstream::app;
	const int nNAgent = 6;
	const int nNLine = 6;
	const int nSimu = 100;
	const int nRepet = 10;
	int nAgent[nNAgent] = { 10, 100, 500, 1000, 5000, 10000 };//, 10000

	int nLine[nNLine] = { 10, 100, 500, 1000, 5000, 10000 }; // 10000 };
	int blockSize = 256;
	float values[nSimu];
	
	MatrixCPU temps(nNAgent*nNLine, nSimu, 0);
	MatrixCPU nAgentMat(1, nNAgent);
	MatrixCPU nLineMat(1, nNLine);
	for (int j = 0; j < nSimu; j++) {
		values[j] = (float)(rand()) / rand();
	}
	int indice = 0;
	for (int i = 0; i < nNAgent; i++) {
		nAgentMat.set(0, i, nAgent[i]);
		for (int j = 0; j < nNLine; j++) {
			nLineMat.set(0, j, nLine[j]);
			std::cout << "iteration (" << i << ", " << j << ") nAgent " << nAgent[i] << " nline " << nLine[j] << std::endl;
			for (int simu = 0; simu < nSimu; simu++) {
				MatrixGPU alpha(nLine[j], nAgent[i], values[j], 1);
				MatrixGPU Qpart(nLine[j], nAgent[i], 0, 1);
				MatrixGPU alphaTrans(nAgent[i], nLine[j], values[j], 1);
				MatrixGPU QpartTrans(nAgent[i], nLine[j], 0, 1);
				//alpha.setRand(values[j]);
				//alphaTrans.setTrans(&alpha);
				hipDeviceSynchronize();
				time = 0;
				int numBlocks;
				for (int repet = 0; repet < nRepet; repet++) {
					MatrixGPU alphaCopy(alpha);
					MatrixGPU QpartCopy(Qpart);
					MatrixGPU alphaTransCopy(alphaTrans);
					MatrixGPU QpartTransCopy(QpartTrans);
					int N = nAgent[i];
					int L = nLine[j];
					hipDeviceSynchronize();
					switch (method)
					{
					case 0:
						numBlocks = L;
						a = std::chrono::high_resolution_clock::now();
						calculQpartLineBloc <<<numBlocks, blockSize, N * sizeof(float) >> > (QpartCopy._matrixGPU, alphaCopy._matrixGPU, N);
						hipDeviceSynchronize();
						b = std::chrono::high_resolution_clock::now();
						break;
					case 1:
						numBlocks = N;
						a = std::chrono::high_resolution_clock::now();
						calculQpartAgentBloc <<<numBlocks, blockSize >> > (QpartCopy._matrixGPU, alphaCopy._matrixGPU, L, N);
						hipDeviceSynchronize();
						b = std::chrono::high_resolution_clock::now();
						break;
					case 2:
						numBlocks = L;
						a = std::chrono::high_resolution_clock::now();
						calculQpartLineBlocTrans << <numBlocks, blockSize, N * sizeof(float) >> > (QpartTransCopy._matrixGPU, alphaTransCopy._matrixGPU, N, L);
						hipDeviceSynchronize();
						b = std::chrono::high_resolution_clock::now();
						break;
					case 3: 
						numBlocks = N;
						a = std::chrono::high_resolution_clock::now();
						calculQpartAgentBlocTrans << <numBlocks, blockSize >> > (QpartTransCopy._matrixGPU, alphaTransCopy._matrixGPU, L, N);
						hipDeviceSynchronize();
						b = std::chrono::high_resolution_clock::now();
						break;
					case 4:
						numBlocks = L;
						a = std::chrono::high_resolution_clock::now();
						calculQpartLineBlocReverse << <numBlocks, blockSize, N * sizeof(float) >> > (QpartCopy._matrixGPU, alphaCopy._matrixGPU, N);
						hipDeviceSynchronize();
						b = std::chrono::high_resolution_clock::now();
						break;
					case 5:
						numBlocks = L;
						a = std::chrono::high_resolution_clock::now();
						calculQpartLineBlocReverseTrans << <numBlocks, blockSize, N * sizeof(float) >> > (QpartTransCopy._matrixGPU, alphaTransCopy._matrixGPU, N, L);
						hipDeviceSynchronize();
						b = std::chrono::high_resolution_clock::now();
						break;
					case 6:
						numBlocks = L;
						a = std::chrono::high_resolution_clock::now();
						calculQpartLineBlocReverseBis << <numBlocks, blockSize, N * sizeof(float) >> > (QpartCopy._matrixGPU, alphaCopy._matrixGPU, N);
						hipDeviceSynchronize();
						b = std::chrono::high_resolution_clock::now();
						break;
					case 7:
						numBlocks = L;
						a = std::chrono::high_resolution_clock::now();
						calculQpartLineBlocReverseBisTrans << <numBlocks, blockSize, N * sizeof(float) >> > (QpartTransCopy._matrixGPU, alphaTransCopy._matrixGPU, N, L);
						hipDeviceSynchronize();
						b = std::chrono::high_resolution_clock::now();
						break;
					default:
						return 0;
						break;
					}
					time += std::chrono::duration_cast<std::chrono::nanoseconds>(b - a).count();
				}
				temps.set(indice, simu, (float)time / nRepet);
			}
			indice++;
		}
	}
	nAgentMat.saveCSV(fileName, mode);
	nLineMat.saveCSV(fileName, mode);
	temps.saveCSV(fileName, mode);
	return temps.sum();
}

__global__ void calculQpartLineBloc(float* Qpart, float* alpha, const int N)
{
	int index = threadIdx.x;
	int step = blockDim.x;
	int l = blockIdx.x;
	extern __shared__ float shAlpha[];

	for (int n = index; n < N; n += step)
	{
		shAlpha[n] = alpha[l * N + n];
	}
	__syncthreads();

	for (int n = index; n < N; n += step)
	{
		float s = 0;
		for (int j = n + 1; j < N; j++) {
			s += shAlpha[j]; // c'est moche cet accès de mémoire partagée
		}
		Qpart[l*N + n] = s;
	}
}

__global__ void calculQpartAgentBloc(float* Qpart, float* alpha, const int L, const int N) {
	int index = threadIdx.x;
	int step = blockDim.x;
	int n = blockIdx.x;
	

	for (int l = index; l < L; l += step)
	{
		float s = 0;
		for (int j = n + 1; j < N; j++) {
			s += alpha[l*N+j]; 
		}
		Qpart[l*N + n] = s;
	}
}

__global__ void calculQpartLineBlocTrans(float* Qpart, float* alpha, const int N, const int nLine)
{
	int index = threadIdx.x;
	int step = blockDim.x;
	int l = blockIdx.x;
	extern __shared__ float shAlpha[];

	for (int n = index; n < N; n += step)
	{
		shAlpha[n] = alpha[ n*nLine + l]; // moche
	}
	__syncthreads();

	for (int n = index; n < N; n += step)
	{
		float s = 0;
		for (int j = n + 1; j < N; j++) {
			s += shAlpha[j]; // c'est moche cet accès de mémoire partagée
		}
		Qpart[ n*nLine +l] = s; // moche
	}
}

__global__ void calculQpartLineBlocReverseTrans(float* Qpart, float* alpha, const int N, const int nLine)
{
	int index = threadIdx.x;
	int step = blockDim.x;
	int l = blockIdx.x;
	extern __shared__ float shAlpha[];

	for (int n = index; n < N; n += step)
	{
		shAlpha[n] = alpha[n * nLine + l]; // moche
	}
	__syncthreads();
	float s_pre = 0;
	int n_pre = N - 1;
	for (int n = (N - index - 1); n >= 0; n -= step)
	{
		float s = 0;
		for (int j = n_pre; j > n; j--) {
			s += shAlpha[j]; // c'est moche cet accès de mémoire partagée
		}
		s = s + s_pre;
		Qpart[n * nLine + l] = s;
		s_pre = s;
		n_pre = n;
	}
}

__global__ void calculQpartAgentBlocTrans(float* Qpart, float* alpha, const int L, const int N) {
	int index = threadIdx.x;
	int step = blockDim.x;
	int n = blockIdx.x;
	

	for (int l = index; l < L; l += step)
	{
		float s = 0;
		for (int j = n + 1; j < N; j++) {
			s += alpha[j * L + l]; // nombre de boucle dépend du bloc pas du thread, accès coalescent
		}
		Qpart[ n * L + l] = s;
	}
}

__global__ void calculQpartLineBlocReverse(float* Qpart, float* alpha, const int N) // est ce que cela marche ???????
{
	int index = threadIdx.x;
	int step = blockDim.x;
	int l = blockIdx.x;
	extern __shared__ float shAlpha[];

	for (int n = index; n < N; n += step)
	{
		shAlpha[n] = alpha[l * N + n];
	}
	__syncthreads();

	float s_pre = 0;
	int n_pre = N-1;
	for (int n = (N-index-1); n >=0; n -= step)
	{
		float s = 0;
		for (int j = n_pre; j > n; j--) {
			s += shAlpha[j]; // c'est moche cet accès de mémoire partagée
		}
		s = s + s_pre;
		Qpart[l * N + n] = s; 
		s_pre = s;
		n_pre = n;
	}
}

__global__ void calculQpartLineBlocReverseBis(float* Qpart, float* alpha, const int N) // est ce que cela marche ???????
{
	int index = threadIdx.x;
	int step = blockDim.x;
	int l = blockIdx.x;
	extern __shared__ float shAlpha[];

	for (int n = index; n < N; n += step)
	{
		shAlpha[n] = alpha[l * N + n];
	}
	__syncthreads();

	for (int n = (N - index - 1); n >= 0; n -= step)
	{
		float s = 0;
		for (int j = N-1; j > n; j--) {
			s += shAlpha[j]; // c'est moche cet accès de mémoire partagée
		}
		Qpart[l * N + n] = s;
	}
}

__global__ void calculQpartLineBlocReverseBisTrans(float* Qpart, float* alpha, const int N, const int nLine) // est ce que cela marche ???????
{
	int index = threadIdx.x;
	int step = blockDim.x;
	int l = blockIdx.x;
	extern __shared__ float shAlpha[];

	for (int n = index; n < N; n += step)
	{
		shAlpha[n] = alpha[l * N + n];
	}
	__syncthreads();

	for (int n = (N - index - 1); n >= 0; n -= step)
	{
		float s = 0;
		for (int j = N - 1; j > n; j--) {
			s += shAlpha[j]; // c'est moche cet accès de mémoire partagée
		}
		Qpart[n * nLine + l] = s;
	}
}

// --------------------------------- alpha -------------------------------------------------------

float testCalculAlpha(int method)
{
	std::string fileName = "TempsAlpha" + std::to_string(method) + ".csv";
	//float elapsedTime;
	std::chrono::high_resolution_clock::time_point a;
	std::chrono::high_resolution_clock::time_point b;
	unsigned int time;
	std::ios_base::openmode mode = std::fstream::in | std::fstream::out | std::fstream::app;
	const int nNAgent = 6;
	const int nNLine = 6;
	const int nSimu = 100;
	const int nRepet = 10;
	int nAgent[nNAgent] = { 10, 100, 500, 1000, 5000, 10000 };
	int nLine[nNLine] = { 10, 100, 500, 1000, 5000, 10000 };
	int blockSize = 256;
	float values[nSimu];
	float values2[nSimu];

	MatrixCPU temps(nNAgent * nNLine, nSimu, 0);
	MatrixCPU nAgentMat(1, nNAgent);
	MatrixCPU nLineMat(1, nNLine);
	for (int j = 0; j < nSimu; j++) {
		values[j] = (float)(rand()) / rand();
		values2[j] = (float)(rand()) / rand();
	}
	int indice = 0;
	for (int i = 0; i < nNAgent; i++) {
		nAgentMat.set(0, i, nAgent[i]);
		for (int j = 0; j < nNLine; j++) {
			nLineMat.set(0, j, nLine[j]);
			//std::cout << "iteration (" << i << ", " << j << ") nAgent " << nAgent[i] << " nline " << nLine[j] << std::endl;
			for (int simu = 0; simu < nSimu; simu++) {
				MatrixGPU G(nLine[j], nAgent[i], values[j], 1);
				MatrixGPU Pn(nAgent[i], 1, values2[j], 1);
				MatrixGPU GTrans(nAgent[i], nLine[j], values[j], 1);
				MatrixGPU alpha(nLine[j], nAgent[i], 0, 1);
				MatrixGPU alphaTrans(nAgent[i], nLine[j], 0, 1);
				//G.setRand(values[j]);
				//GTrans.setTrans(&G);
				//Pn.setRand(values2[j]);
				hipDeviceSynchronize();
				time = 0;
				int numBlocks;
				
				for (int repet = 0; repet < nRepet; repet++) {
					MatrixGPU alphaCopy(alpha);
					MatrixGPU PnCopy(Pn);
					MatrixGPU GCopy(G);
					MatrixGPU alphaTransCopy(alphaTrans);
					MatrixGPU GTransCopy(GTrans);
					int N = nAgent[i];
					int L = nLine[j];
					const int nThread = 16;
					const int bx = (N + nThread - 1) / nThread;
					const int by = (L + nThread - 1) / nThread;
					dim3 dimBlock(nThread, nThread);
					dim3 gridBlock(bx, by);
					dim3 gridBlockTrans(by, bx);
				
					hipDeviceSynchronize();
					switch (method)
					{
					case 0:
						numBlocks = N;
						a = std::chrono::high_resolution_clock::now();
						updateAlphaSh << <numBlocks, blockSize >> > (alphaCopy._matrixGPU, GCopy._matrixGPU, PnCopy._matrixGPU, L, N);
						hipDeviceSynchronize();
						b = std::chrono::high_resolution_clock::now();
						break;
					case 1:
						a = std::chrono::high_resolution_clock::now();
						updateAlpha2D << <gridBlock, dimBlock >> > (alphaCopy._matrixGPU, GCopy._matrixGPU, PnCopy._matrixGPU, L, N);
						hipDeviceSynchronize();
						b = std::chrono::high_resolution_clock::now();
						break;
					case 2:
						numBlocks = (N * L + blockSize - 1) / blockSize;
						a = std::chrono::high_resolution_clock::now();
						updateAlpha1D << <numBlocks, blockSize >> > (alphaCopy._matrixGPU, GCopy._matrixGPU, PnCopy._matrixGPU, L, N);
						hipDeviceSynchronize();
						b = std::chrono::high_resolution_clock::now();
						break;
					case 3: 
						numBlocks = N;
						a = std::chrono::high_resolution_clock::now();
						updateAlphaShTrans << <numBlocks, blockSize >> > (alphaTransCopy._matrixGPU, GTransCopy._matrixGPU, PnCopy._matrixGPU, L, N);
						hipDeviceSynchronize();
						b = std::chrono::high_resolution_clock::now();
						break;
					case 4: 
						a = std::chrono::high_resolution_clock::now();
						updateAlpha2DTrans << <gridBlockTrans, dimBlock >> > (alphaTransCopy._matrixGPU, GTransCopy._matrixGPU, PnCopy._matrixGPU, L, N);
						hipDeviceSynchronize();
						b = std::chrono::high_resolution_clock::now();
						break;
					case 5: 
						numBlocks = (N * L + blockSize - 1) / blockSize;
						a = std::chrono::high_resolution_clock::now();
						updateAlpha1DTrans << <numBlocks, blockSize >> > (alphaTransCopy._matrixGPU, GTransCopy._matrixGPU, PnCopy._matrixGPU, L, N);
						hipDeviceSynchronize();
						b = std::chrono::high_resolution_clock::now();
						break;
						
					default:
						return 0;
						break;
					}
					time += std::chrono::duration_cast<std::chrono::nanoseconds>(b - a).count();
				}
				temps.set(indice, simu, (float)time / nRepet);
			}
			indice++;
		}
	}
	nAgentMat.saveCSV(fileName, mode);
	nLineMat.saveCSV(fileName, mode);
	temps.saveCSV(fileName, mode);
	return temps.sum();
}


__global__ void updateAlphaSh(float* alpha, float* G, float* Pn, const int nLine, const int nAgent)
{
	// un bloc par agent
	int index = threadIdx.x;
	int step = blockDim.x;
	int n = blockIdx.x;
	__shared__ float shPn;
	if (index == 0) {
		shPn = Pn[n];
	}
	__syncthreads();
	for (int l = index; l < nLine; l += step)
	{
		alpha[l * nAgent + n] = G[l * nAgent + n] * shPn;
	}
}

__global__ void updateAlpha2D(float* alpha, float* G, float* Pn, const int nLine, const int nAgent)
{
	int indexX = threadIdx.x + blockIdx.x*blockDim.x;
	int stepX = blockDim.x*gridDim.x;
	int indexY = threadIdx.y + blockIdx.y * blockDim.y;
	int stepY = blockDim.y * gridDim.y;
	
	for (int n = indexX; n < nAgent; n += stepX)
	{
		float PnLocal = Pn[n];
		for (int l = indexY; l < nLine; l += stepY)
		{
			alpha[l * nAgent + n] = G[l * nAgent + n] * PnLocal;
		}
	}
}

__global__ void updateAlpha1D(float* alpha, float* G, float* Pn, const int nLine, const int nAgent)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	int step = blockDim.x * gridDim.x;
	int N = nAgent * nLine;

	for (int i = index; i < N; i += step)
	{
		int k = i % nAgent;
		alpha[i] = G[i] * Pn[k];
	}
}

__global__ void updateAlphaShTrans(float* alpha, float* G, float* Pn, const int nLine, const int nAgent)
{
	// un bloc par agent
	int index = threadIdx.x;
	int step = blockDim.x;
	int n = blockIdx.x;
	__shared__ float shPn;
	if (index == 0) {
		shPn = Pn[n];
	}
	__syncthreads();
	for (int l = index; l < nLine; l += step)
	{
		alpha[ n * nLine + l ] = G[n * nLine + l] * shPn;
	}
}

__global__ void updateAlpha2DTrans(float* alpha, float* G, float* Pn, const int nLine, const int nAgent)
{
	// alpha et G en (n,l)
	int indexX = threadIdx.x + blockIdx.x * blockDim.x;
	int stepX = blockDim.x * gridDim.x;
	int indexY = threadIdx.y + blockIdx.y * blockDim.y;
	int stepY = blockDim.y * gridDim.y;

	for (int n = indexY; n < nAgent; n += stepY)
	{
		float PnLocal = Pn[n];
		for (int l = indexX; l < nLine; l += stepX)
		{
			alpha[n*nLine +l] = G[n * nLine + l] * PnLocal;
		}
	}
}

__global__ void updateAlpha1DTrans(float* alpha, float* G, float* Pn, const int nLine, const int nAgent) {

	int index = threadIdx.x + blockIdx.x * blockDim.x;
	int step = blockDim.x * gridDim.x;
	int N = nAgent * nLine;

	for (int i = index; i < N; i += step)
	{
		int k = i / nAgent;
		alpha[i] = G[i] * Pn[k];
	}

}

// --------------------------------- Cpa -------------------------------------------------------

float testCalculCpa(int method) {
	std::string fileName = "TempsCpa" + std::to_string(method) + ".csv";
	//float elapsedTime;
	std::chrono::high_resolution_clock::time_point a;
	std::chrono::high_resolution_clock::time_point b;
	unsigned int time;
	std::ios_base::openmode mode = std::fstream::in | std::fstream::out | std::fstream::app;
	const int nNAgent = 6;
	const int nNLine = 6;
	const int nSimu = 100;
	const int nRepet = 10;
	int nAgent[nNAgent] = { 10, 100, 500, 1000, 5000, 10000 };
	int nLine[nNLine] = { 10, 100, 500, 1000, 5000, 10000 };
	int blockSize = 256;
	float values[nSimu];
	float values2[nSimu];

	MatrixCPU temps(nNAgent * nNLine, nSimu, 0);
	MatrixCPU nAgentMat(1, nNAgent);
	MatrixCPU nLineMat(1, nNLine);
	for (int j = 0; j < nSimu; j++) {
		values[j] = (float)(rand()) / rand();
		values2[j] = (float)(rand()) / rand();
	}
	int indice = 0;
	for (int i = 0; i < nNAgent; i++) {
		nAgentMat.set(0, i, nAgent[i]);
		for (int j = 0; j < nNLine; j++) {
			nLineMat.set(0, j, nLine[j]);
			//std::cout << "iteration (" << i << ", " << j << ") nAgent " << nAgent[i] << " nline " << nLine[j] << std::endl;
			for (int simu = 0; simu < nSimu; simu++) {
				MatrixGPU Cp2(nAgent[i], 1, 0, 1);
				MatrixGPU tempL1(nLine[j], 1, values[j], 1);
				MatrixGPU G(nLine[j], nAgent[i], values2[j], 1);
				MatrixGPU GTrans(nAgent[i], nLine[j], values2[j], 1);
				//tempL1.setRand(values[j]);
				//G.setRand(values2[j]);
				//GTrans.setTrans(&G);
				hipDeviceSynchronize();
				time = 0;
				int numBlocks;
				for (int repet = 0; repet < nRepet; repet++) {
					MatrixGPU Cp2Copy(Cp2);
					MatrixGPU tempL1Copy(tempL1);
					MatrixGPU GCopy(G);
					MatrixGPU GTransCopy(GTrans);
					int N = nAgent[i];
					int L = nLine[j];
					hipDeviceSynchronize();
					switch (method)
					{
					case 0:
						numBlocks = N;
						a = std::chrono::high_resolution_clock::now();
						updateCp2aTest<256> << <numBlocks, blockSize >> > (Cp2Copy._matrixGPU, tempL1Copy._matrixGPU, GCopy._matrixGPU, L, N);
						hipDeviceSynchronize();
						b = std::chrono::high_resolution_clock::now();
						break;
					case 1:
						numBlocks = N;
						a = std::chrono::high_resolution_clock::now();
						updateCp2aTestTrans<256> << <numBlocks, blockSize >> > (Cp2Copy._matrixGPU, tempL1Copy._matrixGPU, GTransCopy._matrixGPU, L, N);
						hipDeviceSynchronize();
						b = std::chrono::high_resolution_clock::now();
						break;
					default:
						return 0;
						break;
					}
					time += std::chrono::duration_cast<std::chrono::nanoseconds>(b - a).count();
				}
				temps.set(indice, simu, (float)time / nRepet);
			}
			indice++;
		}
	}
	nAgentMat.saveCSV(fileName, mode);
	nLineMat.saveCSV(fileName, mode);
	temps.saveCSV(fileName, mode);
	return temps.sum();
}

template <unsigned int blockSize>
__global__ void updateCp2aTest(float* Cp2, float* diffKappa, float* G, const int nLine, const int nAgent) {
	// un bloc par agent
	int thIdx = threadIdx.x;
	int step = blockDim.x;
	int n = blockIdx.x;
	float sum = 0;
	__shared__ float shArr[blockSize];
	for (int j = thIdx; j < nLine; j += step) {

		float t = G[j * nAgent + n] * diffKappa[j];
		sum += t;
	}

	shArr[thIdx] = sum;
	__syncthreads();

	if (blockSize >= 512) { if (thIdx < 256) { shArr[thIdx] += shArr[thIdx + 256]; } __syncthreads(); }
	if (blockSize >= 256) { if (thIdx < 128) { shArr[thIdx] += shArr[thIdx + 128]; } __syncthreads(); }
	if (blockSize >= 128) { if (thIdx < 64) { shArr[thIdx] += shArr[thIdx + 64]; } __syncthreads(); }
	if (thIdx < 32) {
		warpReduceTest<blockSize>(shArr, thIdx);
	}

	if (thIdx == 0) {
		Cp2[n] = shArr[0];
	}
}

template <unsigned int blockSize>
__global__ void updateCp2aTestTrans(float* Cp2, float* diffKappa, float* G, const int nLine, const int nAgent) {
	// un bloc par agent
	int thIdx = threadIdx.x;
	int step = blockDim.x;
	int n = blockIdx.x;
	float sum = 0;
	__shared__ float shArr[blockSize];
	for (int j = thIdx; j < nLine; j += step) {

		float t = G[ n * nLine + j] * diffKappa[j];
		sum += t;
	}

	shArr[thIdx] = sum;
	__syncthreads();

	if (blockSize >= 512) { if (thIdx < 256) { shArr[thIdx] += shArr[thIdx + 256]; } __syncthreads(); }
	if (blockSize >= 256) { if (thIdx < 128) { shArr[thIdx] += shArr[thIdx + 128]; } __syncthreads(); }
	if (blockSize >= 128) { if (thIdx < 64) { shArr[thIdx] += shArr[thIdx + 64]; } __syncthreads(); }
	if (thIdx < 32) {
		warpReduceTest<blockSize>(shArr, thIdx);
	}

	if (thIdx == 0) {
		Cp2[n] = shArr[0];
	}
}

// --------------------------------- Cpb -------------------------------------------------------

float testCalculCpb(int method)
{
	std::string fileName = "TempsCpb" + std::to_string(method) + ".csv";
	//float elapsedTime;
	std::chrono::high_resolution_clock::time_point a;
	std::chrono::high_resolution_clock::time_point b;
	unsigned int time;
	std::ios_base::openmode mode = std::fstream::in | std::fstream::out | std::fstream::app;
	const int nNAgent = 6;
	const int nNLine = 6;
	const int nSimu = 100;
	const int nRepet = 10;
	int nAgent[nNAgent] = { 10, 100, 500, 1000, 5000, 10000 };
	int nLine[nNLine] = { 10, 100, 500, 1000, 5000, 10000 };
	int blockSize = 256;
	float values[nSimu];
	float values2[nSimu];

	MatrixCPU temps(nNAgent * nNLine, nSimu, 0);
	MatrixCPU nAgentMat(1, nNAgent);
	MatrixCPU nLineMat(1, nNLine);
	for (int j = 0; j < nSimu; j++) {
		values[j] = (float)(rand()) / rand();
		values2[j] = (float)(rand()) / rand();
	}
	int indice = 0;
	for (int i = 0; i < nNAgent; i++) {
		nAgentMat.set(0, i, nAgent[i]);
		for (int j = 0; j < nNLine; j++) {
			nLineMat.set(0, j, nLine[j]);
			int N = nAgent[i];
			int L = nLine[j];
			//std::cout << "iteration (" << i << ", " << j << ") nAgent " << nAgent[i] << " nline " << nLine[j] << std::endl;
			for (int simu = 0; simu < nSimu; simu++) {
				MatrixGPU tempN1(N, 1, 0, 1);
				MatrixGPU Qpart(L, N, values[simu], 1);
				MatrixGPU QpartTrans(N, L, values[simu], 1);
				MatrixGPU G(L, N, values2[simu], 1);
				MatrixGPU GTrans(N, L, values2[simu], 1);
				//Qpart.setRand(values[simu]); en faisant de l'aléatoire, la mesure "plante" au bout d'un moment...
				//G.setRand(values2[simu]);
				//GTrans.setTrans(&G);
				//QpartTrans.setTrans(&Qpart);
				hipDeviceSynchronize();
				time = 0;
				int numBlocks;
				for (int repet = 0; repet < nRepet; repet++) {
					MatrixGPU tempN1Copy(tempN1);
					MatrixGPU QpartCopy(Qpart);
					MatrixGPU QpartTransCopy(QpartTrans);
					MatrixGPU GCopy(G);
					MatrixGPU GTransCopy(GTrans);
					
					hipDeviceSynchronize();
					switch (method)
					{
					case 0:
						numBlocks = N;
						a = std::chrono::high_resolution_clock::now();
						updateCp2bTest<256> << <numBlocks, blockSize >> > (tempN1Copy._matrixGPU, GCopy._matrixGPU, QpartCopy._matrixGPU, L, N);
						hipDeviceSynchronize();
						b = std::chrono::high_resolution_clock::now();
						break;
					case 1:
						numBlocks = N;
						a = std::chrono::high_resolution_clock::now();
						updateCp2bTestTrans<256> << <numBlocks, blockSize >> > (tempN1Copy._matrixGPU, GTransCopy._matrixGPU, QpartTransCopy._matrixGPU, L, N);
						hipDeviceSynchronize();
						b = std::chrono::high_resolution_clock::now();
						break;
					default:
						return 0;
						break;
					}
					time += std::chrono::duration_cast<std::chrono::nanoseconds>(b - a).count();
				}
				temps.set(indice, simu, (float)time / nRepet);
			}
			indice++;
		}
	}
	nAgentMat.saveCSV(fileName, mode);
	nLineMat.saveCSV(fileName, mode);
	temps.saveCSV(fileName, mode);
	return temps.sum();
}
template <unsigned int blockSize>
__global__ void updateCp2bTest(float* tempN1, float* G, float* Qpart, const int nLine, const int nAgent)
{
	// un bloc par agent
	int thIdx = threadIdx.x;
	int step = blockDim.x;
	int n = blockIdx.x;
	float sum = 0;
	__shared__ float shArr[blockSize];
	for (int j = thIdx; j < nLine; j += step) {

		float t = G[j * nAgent + n] * Qpart[j * nAgent + n];
		sum += t;
	}

	shArr[thIdx] = sum;
	__syncthreads();

	if (blockSize >= 512) { if (thIdx < 256) { shArr[thIdx] += shArr[thIdx + 256]; } __syncthreads(); }
	if (blockSize >= 256) { if (thIdx < 128) { shArr[thIdx] += shArr[thIdx + 128]; } __syncthreads(); }
	if (blockSize >= 128) { if (thIdx < 64) { shArr[thIdx] += shArr[thIdx + 64]; } __syncthreads(); }
	if (thIdx < 32) {
		warpReduceTest<blockSize>(shArr, thIdx);
	}

	if (thIdx == 0) {
		tempN1[n] = 2 * shArr[0];
	}

}

template <unsigned int blockSize>
__global__ void updateCp2bTestTrans(float* tempN1, float* G, float* Qpart, const int nLine, const int nAgent)
{
	// un bloc par agent
	int thIdx = threadIdx.x;
	int step = blockDim.x;
	int n = blockIdx.x;
	float sum = 0;
	__shared__ float shArr[blockSize];
	for (int j = thIdx; j < nLine; j += step) {

		float t = G[n * nLine + j] * Qpart[n * nLine + j];
		sum += t;
	}

	shArr[thIdx] = sum;
	__syncthreads();

	if (blockSize >= 512) { if (thIdx < 256) { shArr[thIdx] += shArr[thIdx + 256]; } __syncthreads(); }
	if (blockSize >= 256) { if (thIdx < 128) { shArr[thIdx] += shArr[thIdx + 128]; } __syncthreads(); }
	if (blockSize >= 128) { if (thIdx < 64) { shArr[thIdx] += shArr[thIdx + 64]; } __syncthreads(); }
	if (thIdx < 32) {
		warpReduceTest<blockSize>(shArr, thIdx);
	}

	if (thIdx == 0) {
		tempN1[n] = 2 * shArr[0];
	}

}

// --------------------------------- Q tot -------------------------------------------------------

float testCalculQtot(int method)
{
	std::string fileName = "TempsQtot" + std::to_string(method) + ".csv";
	//float elapsedTime;
	std::chrono::high_resolution_clock::time_point a;
	std::chrono::high_resolution_clock::time_point b;
	unsigned int time;
	std::ios_base::openmode mode = std::fstream::in | std::fstream::out | std::fstream::app;
	const int nNAgent = 6;
	const int nNLine = 6;
	const int nSimu = 100;
	const int nRepet = 10;
	int nAgent[nNAgent] = { 10, 100, 500, 1000, 5000, 10000 };
	int nLine[nNLine] = { 10, 100, 500, 1000, 5000, 10000 };
	int blockSize = 256;
	float values[nSimu];
	float values2[nSimu];

	MatrixCPU temps(nNAgent * nNLine, nSimu, 0);
	MatrixCPU nAgentMat(1, nNAgent);
	MatrixCPU nLineMat(1, nNLine);
	for (int j = 0; j < nSimu; j++) {
		values[j] = (float)(rand()) / rand();
		values2[j] = (float)(rand()) / rand();
	}
	int indice = 0;
	for (int i = 0; i < nNAgent; i++) {
		nAgentMat.set(0, i, nAgent[i]);
		for (int j = 0; j < nNLine; j++) {
			nLineMat.set(0, j, nLine[j]);
			//std::cout << "iteration (" << i << ", " << j << ") nAgent " << nAgent[i] << " nline " << nLine[j] << std::endl;
			for (int simu = 0; simu < nSimu; simu++) {
				MatrixGPU Qtot(nLine[j], 1, 0, 1);
				MatrixGPU Qpart(nLine[j], nAgent[i], values[j], 1);
				MatrixGPU QpartTrans(nAgent[i], nLine[j], values[j], 1);
				MatrixGPU alpha(nLine[j], nAgent[i], values2[j], 1);
				MatrixGPU alphaTrans(nAgent[i], nLine[j], values2[j], 1);
				//Qpart.setRand(values[j]);
				//alpha.setRand(values2[j]);
				//QpartTrans.setTrans(&Qpart);
				//alphaTrans.setTrans(&alpha);
				
				hipDeviceSynchronize();
				time = 0;
				int numBlocks;
				for (int repet = 0; repet < nRepet; repet++) {
					MatrixGPU QtotCopy(Qtot);
					MatrixGPU QpartCopy(Qpart);
					MatrixGPU QpartTransCopy(QpartTrans);
					MatrixGPU alphaCopy(alpha);
					MatrixGPU alphaTransCopy(alphaTrans);
					int N = nAgent[i];
					int L = nLine[j];
					hipDeviceSynchronize();
					switch (method)
					{
					case 0:
						a = std::chrono::high_resolution_clock::now();
						QtotCopy.sum(&alphaCopy);
						hipDeviceSynchronize();
						b = std::chrono::high_resolution_clock::now();
						break;
					case 1:
						numBlocks = ( L + blockSize - 1) / blockSize;
						a = std::chrono::high_resolution_clock::now();
						updateQtotTest << <numBlocks, blockSize >> > (QtotCopy._matrixGPU, QpartCopy._matrixGPU, alphaCopy._matrixGPU, L, N);
						hipDeviceSynchronize();
						b = std::chrono::high_resolution_clock::now();
						break;
					case 2:
						numBlocks = (L + blockSize - 1) / blockSize;
						a = std::chrono::high_resolution_clock::now();
						updateQtotTestTrans << <numBlocks, blockSize >> > (QtotCopy._matrixGPU, QpartTransCopy._matrixGPU, alphaTransCopy._matrixGPU, L);
						hipDeviceSynchronize();
						b = std::chrono::high_resolution_clock::now();
						break;
					default:
						return 0;
						break;
					}
					time += std::chrono::duration_cast<std::chrono::nanoseconds>(b - a).count();
				}
				temps.set(indice, simu, (float)time / nRepet);
			}
			indice++;
		}
	}
	nAgentMat.saveCSV(fileName, mode);
	nLineMat.saveCSV(fileName, mode);
	temps.saveCSV(fileName, mode);
	return temps.sum();
}

__global__ void updateQtotTest(float* Qtot, float* Qpart, float* alpha, const int nLine, const int nAgent) {

	
	int thIdx = threadIdx.x + blockIdx.x * blockDim.x;
	int step = blockDim.x * gridDim.x;
	
	for (int l = thIdx; l < nLine; l += step) {

		Qtot[l] = Qpart[l * nAgent] + alpha[l * nAgent];
	}
}

__global__ void updateQtotTestTrans(float* Qtot, float* Qpart, float* alpha, const int nLine) {

	
	int thIdx = threadIdx.x + blockIdx.x * blockDim.x;
	int step = blockDim.x * gridDim.x;

	for (int l = thIdx; l < nLine; l += step) {

		Qtot[l] = Qpart[l] + alpha[l];
	}
}

template <unsigned int blockSize>
__device__ void warpReduceTest(volatile float* sdata, unsigned int tid) {
	if (blockSize >= 64) sdata[tid] += sdata[tid + 32];
	if (blockSize >= 32) sdata[tid] += sdata[tid + 16];
	if (blockSize >= 16) sdata[tid] += sdata[tid + 8];
	if (blockSize >= 8) sdata[tid] += sdata[tid + 4];
	if (blockSize >= 4) sdata[tid] += sdata[tid + 2];
	if (blockSize >= 2) sdata[tid] += sdata[tid + 1];
}

// --------------------------------- Cp -------------------------------------------------------
float testCalculCp(int method) {
	std::string fileName = "TempsCp" + std::to_string(method) + ".csv";
	//float elapsedTime;
	std::chrono::high_resolution_clock::time_point a;
	std::chrono::high_resolution_clock::time_point b;
	unsigned int time;
	std::ios_base::openmode mode = std::fstream::in | std::fstream::out | std::fstream::app;
	const int nNAgent = 6;
	const int nNLine = 6;
	const int nSimu = 100;
	const int nRepet = 10;
	int nAgent[nNAgent] = { 10, 100, 500, 1000, 5000, 10000 };
	int nLine[nNLine] = { 10, 100, 500, 1000, 5000, 10000 };
	int blockSize = 256;
	float values[nSimu];
	float values2[nSimu];
	float values3[nSimu];
	float values4[nSimu];

	MatrixCPU temps(nNAgent * nNLine, nSimu, 0);
	MatrixCPU nAgentMat(1, nNAgent);
	MatrixCPU nLineMat(1, nNLine);
	for (int j = 0; j < nSimu; j++) {
		values[j] = (float)(rand()) / rand();
		values2[j] = (float)(rand()) / rand();
		values3[j] = (float)(rand()) / rand();
		values4[j] = (float)(rand()) / rand();
	}
	int indice = 0;
	for (int i = 0; i < nNAgent; i++) {
		nAgentMat.set(0, i, nAgent[i]);
		for (int j = 0; j < nNLine; j++) {
			nLineMat.set(0, j, nLine[j]);
			std::cout << "iteration (" << i << ", " << j << ") nAgent " << nAgent[i] << " nline " << nLine[j] << std::endl;
			for (int simu = 0; simu < nSimu; simu++) {
				MatrixGPU Cp2(nAgent[i], 1, 0, 1);
				MatrixGPU tempL1(nLine[j], 1, values[j], 1);
				MatrixGPU G(nLine[j], nAgent[i], values2[j], 1);
				MatrixGPU GTrans(nAgent[i], nLine[j], values2[j], 1);
				MatrixGPU Qpart(nLine[j], nAgent[i], values3[j], 1);
				MatrixGPU QpartTrans(nAgent[i], nLine[j], values3[j], 1);
				MatrixGPU nVoisin(nAgent[i], 1, nAgent[i], 1);
				float rho1 = values4[j];
				//tempL1.setRand(values[j]);
				//G.setRand(values2[j]);
				//GTrans.setTrans(&G);
				hipDeviceSynchronize();
				time = 0;
				int numBlocks;
				for (int repet = 0; repet < nRepet; repet++) {
					MatrixGPU Cp2Copy(Cp2);
					MatrixGPU tempL1Copy(tempL1);
					MatrixGPU GCopy(G);
					MatrixGPU GTransCopy(GTrans);
					MatrixGPU QpartCopy(Qpart);
					MatrixGPU QpartTransCopy(QpartTrans);
					MatrixGPU nVoisinCopy(nVoisin);
					int N = nAgent[i];
					int L = nLine[j];
					hipDeviceSynchronize();
					switch (method)
					{
					case 0:
						numBlocks = N;
						a = std::chrono::high_resolution_clock::now();
						updateCp2Test<256> << <numBlocks, blockSize >> > (Cp2Copy._matrixGPU, tempL1Copy._matrixGPU, GCopy._matrixGPU, QpartCopy._matrixGPU, nVoisinCopy._matrixGPU, rho1, L, N);
						hipDeviceSynchronize();
						b = std::chrono::high_resolution_clock::now();
						break;
					case 1:
						numBlocks = N;
						a = std::chrono::high_resolution_clock::now();
						updateCp2TestTrans<256> << <numBlocks, blockSize >> > (Cp2Copy._matrixGPU, tempL1Copy._matrixGPU, GTransCopy._matrixGPU, QpartTransCopy._matrixGPU, nVoisinCopy._matrixGPU, rho1, L, N);
						hipDeviceSynchronize();
						b = std::chrono::high_resolution_clock::now();
						break;
					default:
						return 0;
						break;
					}
					time += std::chrono::duration_cast<std::chrono::nanoseconds>(b - a).count();
				}
				temps.set(indice, simu, (float)time / nRepet);
			}
			indice++;
		}
	}
	nAgentMat.saveCSV(fileName, mode);
	nLineMat.saveCSV(fileName, mode);
	temps.saveCSV(fileName, mode);
	return temps.sum();
}

template <unsigned int blockSize>
__global__ void updateCp2Test(float* Cp2, float* diffKappa, float* G, float* Qpart, float* nVoisin, float rho1, const int nLine, const int nAgent) {
	// un bloc par agent
	int thIdx = threadIdx.x;
	int step = blockDim.x;
	int n = blockIdx.x;
	float sum = 0;
	__shared__ float shArr[blockSize];
	for (int j = thIdx; j < nLine; j += step) {

		float t = G[j * nAgent + n] * (diffKappa[j] + 2 * Qpart[j * nAgent + n]);
		sum += t;
	}

	shArr[thIdx] = sum;
	__syncthreads();

	if (blockSize >= 512) { if (thIdx < 256) { shArr[thIdx] += shArr[thIdx + 256]; } __syncthreads(); }
	if (blockSize >= 256) { if (thIdx < 128) { shArr[thIdx] += shArr[thIdx + 128]; } __syncthreads(); }
	if (blockSize >= 128) { if (thIdx < 64) { shArr[thIdx] += shArr[thIdx + 64]; } __syncthreads(); }
	if (thIdx < 32) {
		warpReduceTest<blockSize>(shArr, thIdx);
	}

	if (thIdx == 0) {
		Cp2[n] = rho1 * nVoisin[n] * shArr[0];
	}
}

template <unsigned int blockSize>
__global__ void updateCp2TestTrans(float* Cp2, float* diffKappa, float* G, float* Qpart,float* nVoisin, float rho1, const int nLine, const int nAgent) {
	// un bloc par agent
	int thIdx = threadIdx.x;
	int step = blockDim.x;
	int n = blockIdx.x;
	float sum = 0;
	__shared__ float shArr[blockSize];
	for (int j = thIdx; j < nLine; j += step) {
		float Gloc = G[n * nLine + j];
		float dKloc = diffKappa[j];
		float Q = Qpart[n * nLine + j];
		float t = Gloc * (dKloc + 2 * Q);
		sum += t;
	}

	shArr[thIdx] = sum;
	__syncthreads();

	if (blockSize >= 512) { if (thIdx < 256) { shArr[thIdx] += shArr[thIdx + 256]; } __syncthreads(); }
	if (blockSize >= 256) { if (thIdx < 128) { shArr[thIdx] += shArr[thIdx + 128]; } __syncthreads(); }
	if (blockSize >= 128) { if (thIdx < 64) { shArr[thIdx] += shArr[thIdx + 64]; } __syncthreads(); }
	if (thIdx < 32) {
		warpReduceTest<blockSize>(shArr, thIdx);
	}

	if (thIdx == 0) {
		Cp2[n] = rho1 * nVoisin[n] * shArr[0];
	}
}

// --------------------------------- RexX -------------------------------------------------------
float testCalculResX(int method)
{
	std::string fileName = "TempsResX" + std::to_string(method) + ".csv";
	//float elapsedTime;
	std::chrono::high_resolution_clock::time_point a;
	std::chrono::high_resolution_clock::time_point b;
	unsigned int time;
	std::ios_base::openmode mode = std::fstream::in | std::fstream::out | std::fstream::app;
	const int nNAgent = 1;
	const int nNLine = 8;
	const int nSimu = 100;
	const int nRepet = 10;
	int nLine[nNLine] = { 10, 100, 500, 1000, 5000, 10000, 100000, 1000000 };
	int blockSize = 256;
	float values[nSimu];
	float values2[nSimu];
	float values3[nSimu];
	float values4[nSimu];

	MatrixCPU temps(nNAgent * nNLine, nSimu, 0);
	MatrixCPU nAgentMat(1, nNAgent);
	MatrixCPU nLineMat(1, nNLine);
	for (int j = 0; j < nSimu; j++) {
		values[j] =  (float)(rand()) / rand() * (0.5 - rand() % 2);
		values2[j] = (float)(rand()) / rand() * (0.5 - rand() % 2);
		values3[j] = (float)(rand()) / rand() * (0.5 - rand() % 2);
		values4[j] = (float)(rand()) / rand() * (0.5 - rand() % 2);
	}
	int indice = 0;
	for (int i = 0; i < nNAgent; i++) {
		for (int j = 0; j < nNLine; j++) {
			nLineMat.set(0, j, nLine[j]);
			int numBlocks = ceil((nLine[j] + blockSize - 1) / blockSize);
			std::cout << "iteration (" << i << ", " << j << ") "  << " nline " << nLine[j] << std::endl;
			for (int simu = 0; simu < nSimu; simu++) {
				MatrixGPU res(nLine[j], 1, 0, 1);
				MatrixGPU tempL2(nLine[j], 1, 0, 1);
				MatrixGPU kappa1(nLine[j], 1, values[j], 1);
				MatrixGPU kappa2(nLine[j], 1, values2[j], 1);
				MatrixGPU kappaPre1(nLine[j], 1, values3[j], 1);
				MatrixGPU kappaPre2(nLine[j], 1, values4[j], 1);
				hipDeviceSynchronize();
				time = 0;
				for (int repet = 0; repet < nRepet; repet++) {
					MatrixGPU resCopy(res);
					MatrixGPU tempL2Copy(tempL2);
					MatrixGPU kappa1Copy(kappa1);
					MatrixGPU kappa2Copy(kappa2);
					MatrixGPU kappaPre1Copy(kappaPre1);
					MatrixGPU kappaPre2Copy(kappaPre2);
					int L = nLine[j];
					hipDeviceSynchronize();
					switch (method)
					{
					case 0:
						a = std::chrono::high_resolution_clock::now();
						resCopy.set(&kappa1Copy);
						tempL2.set(&kappa2Copy);
						kappaPre1Copy.projectNeg();
						kappaPre2Copy.projectNeg();
						resCopy.projectNeg();
						tempL2.projectNeg();
						resCopy.subtract(&kappaPre1Copy);
						tempL2.subtract(&kappaPre2Copy);
						resCopy.multiplyT(&resCopy);
						tempL2.multiplyT(&tempL2);
						resCopy.add(&tempL2);
						hipDeviceSynchronize();
						b = std::chrono::high_resolution_clock::now();
						break;
					case 1:
						a = std::chrono::high_resolution_clock::now();
						updateResXTest << <numBlocks, blockSize >> > (resCopy._matrixGPU, kappa1Copy._matrixGPU, kappa2Copy._matrixGPU, kappaPre1Copy._matrixGPU, kappaPre2Copy._matrixGPU, L);
						hipDeviceSynchronize();
						b = std::chrono::high_resolution_clock::now();
						break;
					default:
						return 0;
						break;
					}
					time += std::chrono::duration_cast<std::chrono::nanoseconds>(b - a).count();
				}
				temps.set(indice, simu, (float)time / nRepet);
			}
			indice++;
		}
	}
	nAgentMat.saveCSV(fileName, mode);
	nLineMat.saveCSV(fileName, mode);
	temps.saveCSV(fileName, mode);
	return temps.sum();
}

__global__ void updateResXTest(float* res, float* Kappa1, float* Kappa2, float* KappaPre1, float* KappaPre2, const int nLine) {

	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int step = blockDim.x * gridDim.x;
	for (int l = index; l < nLine; l += step)
	{
		float k1 = (Kappa1[l] < 0) * Kappa1[l];
		float k2 = (Kappa2[l] < 0) * Kappa2[l];
		float k1pre = (KappaPre1[l] < 0) * KappaPre1[l];
		float k2pre = (KappaPre2[l] < 0) * KappaPre2[l];

		k1 -= k1pre;
		k2 -= k2pre;

		res[l] = k1 * k1 + k2 * k2;	
	}
}

// --------------------------------- LAMBDABt1 -------------------------------------------------------

float testCalculLAMBDABt1(int method)
{
	std::string fileName = "TempsLAMBDA" + std::to_string(method) + ".csv";
	//float elapsedTime;
	std::chrono::high_resolution_clock::time_point a;
	std::chrono::high_resolution_clock::time_point b;
	unsigned int time;
	std::ios_base::openmode mode = std::fstream::in | std::fstream::out | std::fstream::app;
	const int nNAgent = 6;
	const int nSimu = 100;
	const int nRepet = 10;
	int nAgent[nNAgent] = { 10, 100, 500, 1000, 5000, 10000 };
	int ntrade[nNAgent];
	int blockSize = 256;
	float values[nSimu];
	float values2[nSimu];
	float rhos[nSimu];
	MatrixCPU temps(nNAgent, nSimu, 0);
	MatrixCPU nAgentMat(1, nNAgent, 0);
	MatrixCPU nTradeMat(1, nNAgent, 0);
	for (int j = 0; j < nSimu; j++) {
		values[j] = (float)(rand()) / rand();
		values2[j] = (float)(rand()) / rand();
		rhos[j] = (float)(rand() % 100) / rand();
	}
	for (int i = 0; i < nNAgent; i++) {
		ntrade[i] = nAgent[i] * nAgent[i] / 2;
		int numBlocks = ceil((ntrade[i] + blockSize - 1) / blockSize);
		nAgentMat.set(0, i, nAgent[i]);
		nTradeMat.set(0, i, ntrade[i]);
		MatrixGPU Bt1(ntrade[i], 1, 0, 1);
		MatrixGPU CoresLinAgent(ntrade[i], 1);
		MatrixGPU CoresLinVoisin(ntrade[i], 1);
		MatrixGPU CoresMatLin(nAgent[i], nAgent[i], -1);
		MatrixGPU CoresLinTrans(ntrade[i], 1);
		int indice = 0;
		int idVoisin = 0;

		for (int idAgent = 0; idAgent < nAgent[i]; idAgent++) {
			int Nvoisinmax = nAgent[i] / 2;
			if (idAgent < Nvoisinmax) {
				idVoisin = 0;
			}
			else {
				idVoisin = nAgent[i] / 2;
			}
			for (int voisin = idVoisin; voisin < Nvoisinmax; voisin++) {
				CoresLinAgent.set(indice, 0, idAgent);
				CoresLinVoisin.set(indice, 0, voisin);
				CoresMatLin.set(idAgent, voisin, indice);
				indice = indice + 1;
			}
		}
		for (int lin = 0; lin < ntrade[i]; lin++) {
			int i = CoresLinAgent.get(lin, 0);
			int j = CoresLinVoisin.get(lin, 0);
			int k = CoresMatLin.get(j, i);
			CoresLinTrans.set(lin, 0, k);
		}
		CoresLinAgent.transferGPU();
		CoresLinVoisin.transferGPU();
		CoresMatLin.transferGPU();
		CoresLinTrans.transferGPU();
		//std::cout << "iteration (" << i << ", " << j << ") nAgent " << nAgent[i] << " nline " << nLine[j] << std::endl;
		for (int simu = 0; simu < nSimu; simu++) {
			MatrixGPU LAMBDALin(ntrade[i], 1, values[simu], 1);
			MatrixGPU trade(ntrade[i], 1, values2[simu], 1);
			float rho = rhos[simu];
			hipDeviceSynchronize();
			time = 0;
			for (int repet = 0; repet < nRepet; repet++) {
				MatrixGPU LAMBDALinCopy(LAMBDALin);
				MatrixGPU tradeCopy(trade);
				MatrixGPU CoresLinTransCopy(CoresLinTrans);
				MatrixGPU Bt1Copy(Bt1);
				int M = ntrade[i];
				hipDeviceSynchronize();
				switch (method)
				{
				case 0:
					a = std::chrono::high_resolution_clock::now();
					updateLAMBDAGPUTest << <numBlocks, blockSize >> > (LAMBDALinCopy._matrixGPU, tradeCopy._matrixGPU, rho, CoresLinTransCopy._matrixGPU, M);
					updateBt1GPUTest << <numBlocks, blockSize >> > (Bt1Copy._matrixGPU, tradeCopy._matrixGPU, rho, LAMBDALinCopy._matrixGPU, CoresLinTransCopy._matrixGPU, M);
					hipDeviceSynchronize();
					b = std::chrono::high_resolution_clock::now();
					break;
				case 1:
					a = std::chrono::high_resolution_clock::now();
					updateLAMBDABt1GPUTest << <numBlocks, blockSize >> > (Bt1Copy._matrixGPU, LAMBDALinCopy._matrixGPU, tradeCopy._matrixGPU, rho, CoresLinTransCopy._matrixGPU, M);
					hipDeviceSynchronize();
					b = std::chrono::high_resolution_clock::now();
					break;
				default:
					return 0;
					break;
				}
				time += std::chrono::duration_cast<std::chrono::nanoseconds>(b - a).count();
			}
			temps.set(i, simu, (float)time / nRepet);
		}
	}
	nAgentMat.saveCSV(fileName, mode);
	nTradeMat.saveCSV(fileName, mode);
	temps.saveCSV(fileName, mode);
	return temps.sum();
}


__global__ void updateLAMBDAGPUTest(float* LAMBDALin, float* tradeLin, float rho, float* CoresLinTrans, int const N)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int step = blockDim.x * gridDim.x;
	for (int l = index; l < N; l += step)
	{
		float m = LAMBDALin[l];
		int k = CoresLinTrans[l];
		LAMBDALin[l] = m + 0.5 * rho * (tradeLin[l] + tradeLin[k]);
	}
}
__global__ void updateBt1GPUTest(float* Bt1, float* tradeLin, float rho, float* LAMBDA, float* CoresLinTrans, int const N)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int step = blockDim.x * gridDim.x;
	for (int l = index; l < N; l += step)
	{
		int k = CoresLinTrans[l];
		Bt1[l] = 0.5 * (tradeLin[l] - tradeLin[k]) - LAMBDA[l] / rho;
	}

}

__global__ void updateLAMBDABt1GPUTest(float* Bt1, float* LAMBDA, float* tradeLin, float rho, float* CoresLinTrans, int const N) {

	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int step = blockDim.x * gridDim.x;
	for (int l = index; l < N; l += step)
	{
		int k = CoresLinTrans[l];
		float m = LAMBDA[l] + 0.5 * rho * (tradeLin[l] + tradeLin[k]);
		Bt1[l] = 0.5 * (tradeLin[l] - tradeLin[k]) - m / rho;
		LAMBDA[l] = m;
	}


}

// --------------------------------- Chat -------------------------------------------------------
float testCalculChat(int method, int blockSize, int repartition) {
	std::string fileName = "TempsCalculChat" + std::to_string(method) + "_" + std::to_string(blockSize) + "_" + std::to_string(repartition) + ".csv";
	//float elapsedTime;
	std::chrono::high_resolution_clock::time_point a;
	std::chrono::high_resolution_clock::time_point b;
	unsigned int time;
	std::ios_base::openmode mode = std::fstream::in | std::fstream::out | std::fstream::app;
	const int nNBus = 8;
	const int nSimu = 100;
	const int nRepet = 10;
	int nBus[nNBus] = { 10, 100, 500, 1000, 5000, 10000, 50000, 100000 };

	float values[nSimu];
	float values2[nSimu];
	
	float rhos[nSimu];
	MatrixCPU temps(nNBus, nSimu, 0);
	MatrixCPU nBusMat(1, nNBus, 0);
	for (int j = 0; j < nSimu; j++) {
		values[j] = (float)(rand()) / rand();
		values2[j] = (float)(rand()) / rand();
		rhos[j] = (float)(rand() % 100) / rand();
	}
	for (int i = 0; i < nNBus; i++) {
		std::cout << "iteration (" << i << ") nBus " << nBus[i] << std::endl;
		nBusMat.set(0, i, nBus[i]);
		int numBlocksB = ceil((nBus[i] + blockSize - 1) / blockSize);
		int sizeOPFTotal = 10 * nBus[i] - 3;
		MatrixGPU ChatCopy(6, nBus[i], 0, 1);
		MatrixGPU Bpt2Copy(2 * nBus[i], 1, 0, 1);
		MatrixGPU nAgentByBus(nBus[i], 1, 2, 1);
		MatrixGPU nChild(nBus[i], 1);
		MatrixGPU Ancestor(nBus[i], 1);
		MatrixGPU PosChild(nBus[i], 1);
		MatrixGPU Childs(nBus[i], 1);
		MatrixGPU indiceBusBegin(nBus[i], 1);
		MatrixGPU indiceChildBegin(nBus[i], 1);
		int debut;
		switch (repartition)
		{
		case 0: // tous sur 0
			
			nChild.set(0, 0, nBus[i] - 1);
			Ancestor.set(0, 0, -1);
			debut = 3 * (nBus[i] - 1) + 7;
			for (int b = 1; b < nBus[i]; b++) {
				PosChild.set(b, 0, b - 1);
				Childs.set(b, 0, b + 1);
				indiceChildBegin.set(b, 0, nBus[i]);
				indiceBusBegin.set(b, 0, debut);
				debut += 7;
			}
			break;
		case 1:
			// ligne
			debut = 0;
			for (int b = 0; b < nBus[i]; b++) {
				nChild.set(b, 0, 1);
				Ancestor.set(b, 0, b - 1);
				//PosChild.set(b, 0, 0);
				Childs.set(b, 0, b + 1);
				indiceBusBegin.set(b, 0, debut);
				indiceChildBegin.set(b, 0, b);
				debut += 10;
			}
			break;
		default:
			// n enfant par bus
			int nStep = log(nBus[i]) / log(repartition);
			int BusBegin = 0;
			debut = 0;
			Ancestor.set(0, 0, -1);
			for (int n = 0; n < nStep - 1; n++) {
				int nBusStep = pow(repartition, n);
				for (int ancestor = 0; ancestor < nBusStep; ancestor++) {
					int idAncestor = BusBegin + ancestor; // à changer ?
					nChild.set(idAncestor, 0, repartition);
					indiceBusBegin.set(idAncestor, 0, debut);
					indiceChildBegin.set(idAncestor, 0, idAncestor * repartition);
					debut += 3 * repartition + 7;
					for (int b = 0; b < repartition; b++) {
						int idBus = BusBegin + nBusStep + ancestor * repartition + b; // à changer ?
						Ancestor.set(idBus, 0, idAncestor);
						PosChild.set(idBus, 0, b);
					}
				}
				BusBegin += pow(repartition, n);
			}
			

			int nBusRestant = nBus[i] - (pow(repartition, nStep) - 1);
			int nBAncestor = nBusRestant / repartition;
			std::cout << BusBegin << " " << nBusRestant << " " << nBAncestor << std::endl;
			for (int ancestor = 0; ancestor < nBAncestor; ancestor++) {
				int idAncestor = BusBegin + ancestor; // à changer ?
				indiceBusBegin.set(idAncestor, 0, debut);
				nChild.set(idAncestor, 0, repartition);
				indiceChildBegin.set(idAncestor, 0, idAncestor * repartition);
				debut += 3 * repartition + 7;
				for (int b = 0; b < repartition; b++) {
					int idBus = pow(repartition, nStep) - 1 + ancestor * repartition + b; // à changer ?
					Ancestor.set(idBus, 0, idAncestor);
					PosChild.set(idBus, 0, b);
				}
			}
			
			for (int b = BusBegin + nBAncestor; b < nBus[i]; b++) { // le reste n'a pas d'enfant
				indiceBusBegin.set(b, 0, debut);
				debut += 7;
			}

			nBusRestant = nBusRestant % repartition;

			for (int b = 0; b < nBusRestant; b++) {
				int idBus = nBus[i] - nBusRestant + b; // à changer ?
				Ancestor.set(idBus, 0, 0);
				PosChild.set(idBus, 0, b + repartition);
			}

			for (int b = 0; b < nBus[i]; b++) {
				Childs.set(b, 0, b + 1);
			}
			break;
		}
		nChild.transferGPU();
		Ancestor.transferGPU();
		PosChild.transferGPU();
		Childs.transferGPU();
		indiceBusBegin.transferGPU();
		indiceChildBegin.transferGPU();
		

		for (int simu = 0; simu < nSimu; simu++) {
			MatrixGPU Y(sizeOPFTotal, 1, values[simu], 1);
			MatrixGPU Mu(sizeOPFTotal, 1, values2[simu], 1);
			float _rho = rhos[simu];
			hipDeviceSynchronize();
			time = 0;
			for (int repet = 0; repet < nRepet; repet++) {
				MatrixGPU YCopy(Y);
				MatrixGPU MuCopy(Mu);
				MatrixGPU nChildCopy(nChild);
				MatrixGPU AncestorCopy(Ancestor);
				MatrixGPU PosChildCopy(PosChild);
				MatrixGPU ChildsCopy(Childs);
				MatrixGPU indiceBusBeginCopy(indiceBusBegin);
				MatrixGPU indiceChildBeginCopy(indiceChildBegin);
				
				int B = nBus[i];
				int numBlock = B;
				
				hipDeviceSynchronize();
				switch (method)
				{
				case 0:
					a = std::chrono::high_resolution_clock::now();
					
					switch (blockSize) {
					case 512:
						updateChatGPUTest<512> << <numBlock, blockSize >> > (ChatCopy._matrixGPU, YCopy._matrixGPU, MuCopy._matrixGPU, nChildCopy._matrixGPU, AncestorCopy._matrixGPU, PosChildCopy._matrixGPU, ChildsCopy._matrixGPU, indiceBusBeginCopy._matrixGPU, indiceChildBeginCopy._matrixGPU, _rho, B);
						break;
					case 256:
						updateChatGPUTest<256> << <numBlock, blockSize >> > (ChatCopy._matrixGPU, YCopy._matrixGPU, MuCopy._matrixGPU, nChildCopy._matrixGPU, AncestorCopy._matrixGPU, PosChildCopy._matrixGPU, ChildsCopy._matrixGPU, indiceBusBeginCopy._matrixGPU, indiceChildBeginCopy._matrixGPU, _rho, B);
						break;
					case 128:
						updateChatGPUTest<128> << <numBlock, blockSize >> > (ChatCopy._matrixGPU, YCopy._matrixGPU, MuCopy._matrixGPU, nChildCopy._matrixGPU, AncestorCopy._matrixGPU, PosChildCopy._matrixGPU, ChildsCopy._matrixGPU, indiceBusBeginCopy._matrixGPU, indiceChildBeginCopy._matrixGPU, _rho, B);
						break;
					case 64:
						updateChatGPUTest< 64> << <numBlock, blockSize >> > (ChatCopy._matrixGPU, YCopy._matrixGPU, MuCopy._matrixGPU, nChildCopy._matrixGPU, AncestorCopy._matrixGPU, PosChildCopy._matrixGPU, ChildsCopy._matrixGPU, indiceBusBeginCopy._matrixGPU, indiceChildBeginCopy._matrixGPU, _rho, B);
						break;
					case 32:
						updateChatGPUTest< 32> << <numBlock, blockSize >> > (ChatCopy._matrixGPU, YCopy._matrixGPU, MuCopy._matrixGPU, nChildCopy._matrixGPU, AncestorCopy._matrixGPU, PosChildCopy._matrixGPU, ChildsCopy._matrixGPU, indiceBusBeginCopy._matrixGPU, indiceChildBeginCopy._matrixGPU, _rho, B);
						break;
					case 16:
						updateChatGPUTest< 16> << <numBlock, blockSize >> > (ChatCopy._matrixGPU, YCopy._matrixGPU, MuCopy._matrixGPU, nChildCopy._matrixGPU, AncestorCopy._matrixGPU, PosChildCopy._matrixGPU, ChildsCopy._matrixGPU, indiceBusBeginCopy._matrixGPU, indiceChildBeginCopy._matrixGPU, _rho, B);
						break;
					case  8:
						updateChatGPUTest<  8> << <numBlock, blockSize >> > (ChatCopy._matrixGPU, YCopy._matrixGPU, MuCopy._matrixGPU, nChildCopy._matrixGPU, AncestorCopy._matrixGPU, PosChildCopy._matrixGPU, ChildsCopy._matrixGPU, indiceBusBeginCopy._matrixGPU, indiceChildBeginCopy._matrixGPU, _rho, B);
						break;
					case  4:
						updateChatGPUTest<  4> << <numBlock, blockSize >> > (ChatCopy._matrixGPU, YCopy._matrixGPU, MuCopy._matrixGPU, nChildCopy._matrixGPU, AncestorCopy._matrixGPU, PosChildCopy._matrixGPU, ChildsCopy._matrixGPU, indiceBusBeginCopy._matrixGPU, indiceChildBeginCopy._matrixGPU, _rho, B);
						break;
					case  2:
						updateChatGPUTest<  2> << <numBlock, blockSize >> > (ChatCopy._matrixGPU, YCopy._matrixGPU, MuCopy._matrixGPU, nChildCopy._matrixGPU, AncestorCopy._matrixGPU, PosChildCopy._matrixGPU, ChildsCopy._matrixGPU, indiceBusBeginCopy._matrixGPU, indiceChildBeginCopy._matrixGPU, _rho, B);
						break;
					case  1:
						updateChatGPUTest<  1> << <numBlock, blockSize >> > (ChatCopy._matrixGPU, YCopy._matrixGPU, MuCopy._matrixGPU, nChildCopy._matrixGPU, AncestorCopy._matrixGPU, PosChildCopy._matrixGPU, ChildsCopy._matrixGPU, indiceBusBeginCopy._matrixGPU, indiceChildBeginCopy._matrixGPU, _rho, B);
						break;
					}

					updateBpt2Test << < numBlocksB, blockSize >> > (Bpt2Copy._matrixGPU, ChatCopy._matrixGPU, nAgentByBus._matrixGPU, B);

					
					hipDeviceSynchronize();
					b = std::chrono::high_resolution_clock::now();
					break;
				case 1:
					a = std::chrono::high_resolution_clock::now();
					switch (blockSize) {
					case 512:
						updateChatBpt2Test<512> << <numBlock, blockSize >> > (ChatCopy._matrixGPU, Bpt2Copy._matrixGPU, YCopy._matrixGPU, MuCopy._matrixGPU, nChildCopy._matrixGPU, AncestorCopy._matrixGPU, PosChildCopy._matrixGPU, ChildsCopy._matrixGPU, indiceBusBeginCopy._matrixGPU, indiceChildBeginCopy._matrixGPU, nAgentByBus._matrixGPU, _rho, B);
						break;
					case 256:
						updateChatBpt2Test<256> << <numBlock, blockSize >> > (ChatCopy._matrixGPU, Bpt2Copy._matrixGPU, YCopy._matrixGPU, MuCopy._matrixGPU, nChildCopy._matrixGPU, AncestorCopy._matrixGPU, PosChildCopy._matrixGPU, ChildsCopy._matrixGPU, indiceBusBeginCopy._matrixGPU, indiceChildBeginCopy._matrixGPU, nAgentByBus._matrixGPU, _rho, B);
						break;
					case 128:
						updateChatBpt2Test<128> << <numBlock, blockSize >> > (ChatCopy._matrixGPU, Bpt2Copy._matrixGPU, YCopy._matrixGPU, MuCopy._matrixGPU, nChildCopy._matrixGPU, AncestorCopy._matrixGPU, PosChildCopy._matrixGPU, ChildsCopy._matrixGPU, indiceBusBeginCopy._matrixGPU, indiceChildBeginCopy._matrixGPU, nAgentByBus._matrixGPU, _rho, B);
						break;
					case 64:
						updateChatBpt2Test< 64> << <numBlock, blockSize >> > (ChatCopy._matrixGPU, Bpt2Copy._matrixGPU, YCopy._matrixGPU, MuCopy._matrixGPU, nChildCopy._matrixGPU, AncestorCopy._matrixGPU, PosChildCopy._matrixGPU, ChildsCopy._matrixGPU, indiceBusBeginCopy._matrixGPU, indiceChildBeginCopy._matrixGPU, nAgentByBus._matrixGPU, _rho, B);
						break;
					case 32:
						updateChatBpt2Test< 32> << <numBlock, blockSize >> > (ChatCopy._matrixGPU, Bpt2Copy._matrixGPU, YCopy._matrixGPU, MuCopy._matrixGPU, nChildCopy._matrixGPU, AncestorCopy._matrixGPU, PosChildCopy._matrixGPU, ChildsCopy._matrixGPU, indiceBusBeginCopy._matrixGPU, indiceChildBeginCopy._matrixGPU, nAgentByBus._matrixGPU, _rho, B);
						break;
					case 16:
						updateChatBpt2Test< 16> << <numBlock, blockSize >> > (ChatCopy._matrixGPU, Bpt2Copy._matrixGPU, YCopy._matrixGPU, MuCopy._matrixGPU, nChildCopy._matrixGPU, AncestorCopy._matrixGPU, PosChildCopy._matrixGPU, ChildsCopy._matrixGPU, indiceBusBeginCopy._matrixGPU, indiceChildBeginCopy._matrixGPU, nAgentByBus._matrixGPU, _rho, B);
						break;
					case  8:
						updateChatBpt2Test<  8> << <numBlock, blockSize >> > (ChatCopy._matrixGPU, Bpt2Copy._matrixGPU, YCopy._matrixGPU, MuCopy._matrixGPU, nChildCopy._matrixGPU, AncestorCopy._matrixGPU, PosChildCopy._matrixGPU, ChildsCopy._matrixGPU, indiceBusBeginCopy._matrixGPU, indiceChildBeginCopy._matrixGPU, nAgentByBus._matrixGPU, _rho, B);
						break;
					case  4:
						updateChatBpt2Test<  4> << <numBlock, blockSize >> > (ChatCopy._matrixGPU, Bpt2Copy._matrixGPU, YCopy._matrixGPU, MuCopy._matrixGPU, nChildCopy._matrixGPU, AncestorCopy._matrixGPU, PosChildCopy._matrixGPU, ChildsCopy._matrixGPU, indiceBusBeginCopy._matrixGPU, indiceChildBeginCopy._matrixGPU, nAgentByBus._matrixGPU, _rho, B);
						break;
					case  2:
						updateChatBpt2Test<  2> << <numBlock, blockSize >> > (ChatCopy._matrixGPU, Bpt2Copy._matrixGPU, YCopy._matrixGPU, MuCopy._matrixGPU, nChildCopy._matrixGPU, AncestorCopy._matrixGPU, PosChildCopy._matrixGPU, ChildsCopy._matrixGPU, indiceBusBeginCopy._matrixGPU, indiceChildBeginCopy._matrixGPU, nAgentByBus._matrixGPU, _rho, B);
						break;
					case  1:
						updateChatBpt2Test<  1> << <numBlock, blockSize >> > (ChatCopy._matrixGPU, Bpt2Copy._matrixGPU, YCopy._matrixGPU, MuCopy._matrixGPU, nChildCopy._matrixGPU, AncestorCopy._matrixGPU, PosChildCopy._matrixGPU, ChildsCopy._matrixGPU, indiceBusBeginCopy._matrixGPU, indiceChildBeginCopy._matrixGPU, nAgentByBus._matrixGPU, _rho, B);
						break;
					}
					hipDeviceSynchronize();
					b = std::chrono::high_resolution_clock::now();
					break;
				case 2:
					a = std::chrono::high_resolution_clock::now();
					updateChatBpt2OneDoAll <<< numBlocksB, blockSize >> > (ChatCopy._matrixGPU, Bpt2Copy._matrixGPU, YCopy._matrixGPU, MuCopy._matrixGPU, nChildCopy._matrixGPU, AncestorCopy._matrixGPU, PosChildCopy._matrixGPU, ChildsCopy._matrixGPU, indiceBusBeginCopy._matrixGPU, indiceChildBeginCopy._matrixGPU, nAgentByBus._matrixGPU, _rho, B);
					hipDeviceSynchronize();
					b = std::chrono::high_resolution_clock::now();
					break;
				default:
					return 0;
					break;
				}
				time += std::chrono::duration_cast<std::chrono::nanoseconds>(b - a).count();
			}
			temps.set(i, simu, (float)time / nRepet);
		}
	}
	nBusMat.saveCSV(fileName, mode);
	temps.saveCSV(fileName, mode);
	return temps.sum();
}


template <unsigned int blockSize>
__global__ void updateChatGPUTest(float* Chat,  float* Y, float* MU, float* nChild, float* Ancestor, float* posChild, float* Childs, float* indiceBusBegin, float* indiceChildBegin, float _rho, int nBus) {

	int bus = blockIdx.x;
	int index = threadIdx.x;
	int step = blockDim.x;

	__shared__ float shArr[blockSize]; // c'est grand pour pas grand chose...


	int indice = indiceBusBegin[bus];
	int indiceChild = (bus < (nBus - 1)) ? indiceChildBegin[bus] : 0;
	int nb = nChild[bus];
	int Ai = Ancestor[bus];
	int c = posChild[bus];
	float var = 0;

	if (index < 6) {
		//float Phat, Qhat, lhat, phat, qhat;
		var = Y[indice + index] / 2 - MU[indice + index] / (2 * _rho);
		if (bus > 0) {
			if (index < 3) {
				int nAi = nChild[Ai];
				int indiceAncBus = indiceBusBegin[Ai] + 7 + nAi * index + c;
				//var = indiceAncBus;
				var += Y[indiceAncBus] / 2 - MU[indiceAncBus] / (2 * _rho);
			}
		}
	}
	float vhat = 0;
	float muhat = 0;
	for (int i = index; i < nb; i += step) {
		int c = Childs[indiceChild + i];
		int indiceBusChild = indiceBusBegin[c];
		muhat += MU[indiceBusChild + 6]; // pas du tout coalescent
		vhat += Y[indiceBusChild + 6]; // pas du tout coalescent
	}
	shArr[index] = vhat / (nb + 1) - muhat / (_rho * (nb + 1));
	__syncthreads();
	for (int size = blockSize / 2; size > 0; size /= 2) { //uniform
		if (index < size) {
			shArr[index] += shArr[index + size];
		}
		__syncthreads();
	}

	if (index < 6) {
		if (index == 3) {
			var = shArr[0] + Y[indice + 3] / (nb + 1) - MU[indice + 3] / (_rho * (nb + 1)); //shArr[0];
		}
		Chat[index * nBus + bus] = var; // pas coalescent mais bon perdu pour perdu
	}
}
__global__ void updateBpt2Test(float* Bpt2, float* Chat, float* nAgentByBus, int nBus) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int step = blockDim.x * gridDim.x;


	for (int b = index; b < nBus; b += step) {
		int nA = nAgentByBus[b];

		Bpt2[b] = nA > 0 ? 2 * Chat[b + 4 * nBus] / nA : 0; // �criture coalescente et lecture coalescente
		Bpt2[b + nBus] = nA > 0 ? 2 * Chat[b + 5 * nBus] / nA : 0;

	}

}

template <unsigned int blockSize>
__global__ void updateChatBpt2Test(float* Chat, float* Bpt2, float* Y, float* MU, float* nChild, float* Ancestor, float* posChild, float* Childs, float* indiceBusBegin, float* indiceChildBegin, float* nAgentByBus, float _rho, int nBus) {
	int bus = blockIdx.x;
	int index = threadIdx.x;
	int step = blockDim.x;

	__shared__ float shArr[blockSize]; // c'est grand pour pas grand chose...


	int indice = indiceBusBegin[bus];
	int indiceChild = (bus < (nBus - 1)) ? indiceChildBegin[bus] : 0;
	int nb = nChild[bus];
	int Ai = Ancestor[bus];
	int c = posChild[bus];
	int nA = nAgentByBus[bus];
	float var = 0;

	if (index < 6) {
		//float Phat, Qhat, lhat, phat, qhat;
		var = Y[indice + index] / 2 - MU[indice + index] / (2 * _rho);
		if (bus > 0) {
			if (index < 3) {
				int nAi = nChild[Ai];
				int indiceAncBus = indiceBusBegin[Ai] + 7 + nAi * index + c;
				//var = indiceAncBus;
				var += Y[indiceAncBus] / 2 - MU[indiceAncBus] / (2 * _rho);
			}
		}
	}
	float vhat = 0;
	float muhat = 0;
	for (int i = index; i < nb; i += step) {
		int c = Childs[indiceChild + i];
		int indiceBusChild = indiceBusBegin[c];
		muhat += MU[indiceBusChild + 6]; // pas du tout coalescent
		vhat += Y[indiceBusChild + 6]; // pas du tout coalescent
	}
	shArr[index] = vhat / (nb + 1) - muhat / (_rho * (nb + 1));
	__syncthreads();
	for (int size = blockSize / 2; size > 0; size /= 2) { //uniform
		if (index < size) {
			shArr[index] += shArr[index + size];
		}
		__syncthreads();
	}

	if (index < 6) {
		if (index == 3) {
			var = shArr[0] + Y[indice + 3] / (nb + 1) - MU[indice + 3] / (_rho * (nb + 1)); //shArr[0];
		}
		Chat[index * nBus + bus] = var; // pas coalescent mais bon perdu pour perdu
		if (index == 4) {
			Bpt2[bus] = nA > 0 ? 2 * var / nA : 0; // �criture coalescente et lecture coalescente
		} 
		if (index == 5) {
			Bpt2[bus + nBus] = nA > 0 ? 2 * var / nA : 0;
		}
	}
}


__global__ void updateChatBpt2OneDoAll(float* Chat, float* Bpt2, float* Y, float* MU, float* nChild, float* Ancestor, float* posChild, float* Childs, float* indiceBusBegin, float* indiceChildBegin, float* nAgentByBus, float _rho, int nBus) {

	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int step = blockDim.x * gridDim.x;


	for (int bus = index; bus < nBus; bus += step) {
		int indice = indiceBusBegin[bus];
		int indiceChild = (bus < (nBus - 1)) ? indiceChildBegin[bus] : 0;
		int nb = nChild[bus];
		int Ai = Ancestor[bus];
		int c = posChild[bus];
		int nA = nAgentByBus[bus];
		float Phat, Qhat, lhat, phat, qhat, vhat, muhat;
		
		Phat = Y[indice] / 2     - MU[indice] / (2 * _rho);
		Qhat = Y[indice + 1] / 2 - MU[indice + 1] / (2 * _rho);
		lhat = Y[indice + 2] / 2 - MU[indice + 2] / (2 * _rho);
		phat = Y[indice + 4] / 2 - MU[indice + 4] / (2 * _rho);
		qhat = Y[indice + 5] / 2 - MU[indice + 5] / (2 * _rho);
		vhat = 0;
		muhat = 0;
		if (bus > 0) {
			int nAi = nChild[Ai];
			int indiceAncBus = indiceBusBegin[Ai] + 7 + c;
			Phat += Y[indiceAncBus] / 2 - MU[indiceAncBus] / (2 * _rho);
			Qhat += Y[indiceAncBus + nAi] / 2 - MU[indiceAncBus + nAi] / (2 * _rho);
			lhat += Y[indiceAncBus + 2 * nAi] / 2 - MU[indiceAncBus + 2 * nAi] / (2 * _rho);
		}
		for (int i = 0; i < nb; i++) {
			int c = Childs[indiceChild + i];
			int indiceBusChild = indiceBusBegin[c];
			muhat += MU[indiceBusChild + 6]; // pas du tout coalescent
			vhat += Y[indiceBusChild + 6]; // pas du tout coalescent
		}
		vhat = (vhat + Y[indice + 3]) / (nb + 1) - (muhat + MU[indice + 3]) / (_rho * (nb + 1));
		
		// ecriture coalescente 
		Chat[bus] = Phat;
		Chat[1 * nBus + bus] = Qhat;
		Chat[2 * nBus + bus] = lhat;
		Chat[3 * nBus + bus] = vhat;
		Chat[4 * nBus + bus] = phat;
		Chat[5 * nBus + bus] = qhat;
		Bpt2[bus] = nA > 0 ? 2 * phat / nA : 0; 
		Bpt2[bus + nBus] = nA > 0 ? 2 * qhat / nA : 0;
	}

}


// --------------------------------- probleme local -------------------------------------------------------

float testCalculPnShared(int method, int blockSize, int repartition) {
	std::string fileName = "TempsCalculPnShared" + std::to_string(method) + "_" + std::to_string(blockSize) + "_" + std::to_string(repartition) + ".csv";
	//float elapsedTime;
	std::chrono::high_resolution_clock::time_point a;
	std::chrono::high_resolution_clock::time_point b;
	unsigned int time;
	std::ios_base::openmode mode = std::fstream::in | std::fstream::out | std::fstream::app;
	const int nNBus = 7;
	const int nNAgent = 8;
	const int nSimu = 100;
	const int nRepet = 10;
	int nBus[nNBus] = { 20, 100, 500, 1000, 5000, 10000, 50000 };
	float nAgent[nNAgent] = { 0.1, 0.25, 0.5, 0.75, 1, 2, 5, 10 };
	float epsL = -1; 
	int nIterL = 1000; // pour pas que le nombre d'itération varie en fonction de l'aléatoire, mais induit un biais, un bus avec moins d'agents devrait normalement 
	// nécessiter moins d'itérations. Donc pas généralisable au cas où on cherche la convergence...

	float values[nSimu];
	float values2[nSimu];
	float values3[nSimu];
	float rhos[nSimu];
	MatrixCPU temps(nNBus * nNAgent, nSimu, 0);
	MatrixCPU nBusMat(1, nNBus, 0);
	MatrixCPU nAgentMat(1, nNAgent);
	for (int j = 0; j < nSimu; j++) {
		values[j] = (float)(rand()) / rand();
		values2[j] = (float)(rand()) / rand();
		values3[j] = (float)(rand()) / rand();
		rhos[j] = (float)(rand() % 100) / rand();
	}
	for (int i = 0; i < nNBus; i++) {
		nBusMat.set(0, i, nBus[i]);
		for (int j = 0; j < nNAgent; j++) {
			int Agent = nAgent[j] * nBus[i];
			std::cout << "iteration (" << i << "," << j << ") nBus " << nBus[i] << " agent " << Agent << std::endl;
			nAgentMat.set(0, j, nAgent[j]);
			
			MatrixGPU nAgentByBus(nBus[i], 1);
			MatrixGPU CoresSoloBusAgent(nBus[i], 1, -1);
			MatrixGPU CoresAgentBus(Agent, 1);
			MatrixGPU CoresAgentBusBegin(nBus[i], 1);

		
			

			int nAgentByBusQ = Agent / nBus[i];
			int nAgentByBusR = Agent % nBus[i];

			int debut = 0;
			switch (repartition)
			{
			case 0: // tous sur 0
				nAgentByBus.set(0, 0, Agent);
				
				for (int b = 1; b < nBus[i]; b++) {
					CoresAgentBusBegin.set(b, 0, Agent);
				}

				for (int n = 0; n < Agent; n++) {
					CoresAgentBus.set(n, 0, Agent - 1 - n);
				}
			
				break;
			case 1: // equilibré
				std::cout << nAgentByBusQ << " " << nAgentByBusR << std::endl;
				for (int b = 0; b < nBus[i]; b++) {
					int nA = nAgentByBusQ + 1 * (nAgentByBusR > b);
					nAgentByBus.set(b, 0, nA);
					CoresAgentBusBegin.set(b, 0, debut);
					
					if (nA == 1) {
						CoresSoloBusAgent.set(b, 0, debut);
					}
					debut += nA;

					
				}
				for (int n = 0; n < Agent; n++) {
					CoresAgentBus.set(n, 0, Agent - 1 - n);
				}
				break;
			default: // random
			
				break;
			}
			CoresSoloBusAgent.transferGPU();
			CoresAgentBus.transferGPU();
			CoresAgentBusBegin.transferGPU();
			nAgentByBus.transferGPU();

			for (int simu = 0; simu < nSimu; simu++) {
				
				MatrixGPU Ap2(2 * Agent, 1, 0.1, 1);
				MatrixGPU Apt2(2 * Agent, 1, values2[simu], 1);
				MatrixGPU Cp(2 * Agent, 1, values3[simu], 1);
				MatrixGPU Pmin(2 * Agent, 1, values[simu], 1);
				MatrixGPU Pmax(2 * Agent, 1, values[simu] + 1, 1);
				MatrixGPU Apt1(2 * Agent, 1, values3[simu] - values2[simu] , 1);
				MatrixGPU Bpt2(2 * Agent, 1, values3[simu] + values2[simu], 1);
				float _rhol = rhos[simu];
				
				time = 0;
				for (int repet = 0; repet < nRepet; repet++) {
					MatrixGPU Ap2Copy(Ap2);
					MatrixGPU CpCopy(Cp);
					MatrixGPU PminCopy(Pmin);
					MatrixGPU PmaxCopy(Pmax);
					MatrixGPU Apt1Copy(Apt1);
					MatrixGPU Apt2Copy(Apt2);
					MatrixGPU Bpt2Copy(Bpt2);

					MatrixGPU nAgentByBusCopy(nAgentByBus);
					MatrixGPU CoresSoloBusAgentCopy(CoresSoloBusAgent);
					MatrixGPU CoresAgentBusCopy(CoresAgentBus);
					MatrixGPU CoresAgentBusBeginCopy(CoresAgentBusBegin);

					MatrixGPU PnCopy(2 * Agent, 1, 0, 1);
					MatrixGPU PnPreCopy(2 * Agent, 1, 0, 1);
					MatrixGPU PnMoyCopy(2 * nBus[i], 1, 0, 1);
					MatrixGPU PnTildeCopy(2 * nBus[i], 1, 0, 1);
					MatrixGPU MuLCopy(2 * nBus[i], 1, 0, 1);


					int B = nBus[i];
					int numBlocks = B;

					hipDeviceSynchronize();
					switch (method)
					{
					case 0:
						a = std::chrono::high_resolution_clock::now();
					
						switch (blockSize) {
						case 512:
							updatePnPGPUSharedResidualTest<512> << <numBlocks, blockSize >> > (PnCopy._matrixGPU, PnPreCopy._matrixGPU, PnMoyCopy._matrixGPU, PnTildeCopy._matrixGPU, MuLCopy._matrixGPU, nAgentByBusCopy._matrixGPU, _rhol, Ap2Copy._matrixGPU, CpCopy._matrixGPU,
								PminCopy._matrixGPU, PmaxCopy._matrixGPU, Apt1Copy._matrixGPU, Apt2Copy._matrixGPU, Bpt2Copy._matrixGPU, CoresSoloBusAgentCopy._matrixGPU, CoresAgentBusCopy._matrixGPU, CoresAgentBusBeginCopy._matrixGPU, epsL, nIterL, Agent, B);
							break;
						case 256:
							updatePnPGPUSharedResidualTest<256> << <numBlocks, blockSize >> > (PnCopy._matrixGPU, PnPreCopy._matrixGPU, PnMoyCopy._matrixGPU, PnTildeCopy._matrixGPU, MuLCopy._matrixGPU, nAgentByBusCopy._matrixGPU, _rhol, Ap2Copy._matrixGPU, CpCopy._matrixGPU,
								PminCopy._matrixGPU, PmaxCopy._matrixGPU, Apt1Copy._matrixGPU, Apt2Copy._matrixGPU, Bpt2Copy._matrixGPU, CoresSoloBusAgentCopy._matrixGPU, CoresAgentBusCopy._matrixGPU, CoresAgentBusBeginCopy._matrixGPU, epsL, nIterL, Agent, B);							break;
						case 128:
							updatePnPGPUSharedResidualTest<128> << <numBlocks, blockSize >> > (PnCopy._matrixGPU, PnPreCopy._matrixGPU, PnMoyCopy._matrixGPU, PnTildeCopy._matrixGPU, MuLCopy._matrixGPU, nAgentByBusCopy._matrixGPU, _rhol, Ap2Copy._matrixGPU, CpCopy._matrixGPU,
								PminCopy._matrixGPU, PmaxCopy._matrixGPU, Apt1Copy._matrixGPU, Apt2Copy._matrixGPU, Bpt2Copy._matrixGPU, CoresSoloBusAgentCopy._matrixGPU, CoresAgentBusCopy._matrixGPU, CoresAgentBusBeginCopy._matrixGPU, epsL, nIterL, Agent, B);						break;
						case 64:
							updatePnPGPUSharedResidualTest< 64> << <numBlocks, blockSize >> > (PnCopy._matrixGPU, PnPreCopy._matrixGPU, PnMoyCopy._matrixGPU, PnTildeCopy._matrixGPU, MuLCopy._matrixGPU, nAgentByBusCopy._matrixGPU, _rhol, Ap2Copy._matrixGPU, CpCopy._matrixGPU,
								PminCopy._matrixGPU, PmaxCopy._matrixGPU, Apt1Copy._matrixGPU, Apt2Copy._matrixGPU, Bpt2Copy._matrixGPU, CoresSoloBusAgentCopy._matrixGPU, CoresAgentBusCopy._matrixGPU, CoresAgentBusBeginCopy._matrixGPU, epsL, nIterL, Agent, B);
							break;
						case 32:
							updatePnPGPUSharedResidualTest< 32> << <numBlocks, blockSize >> > (PnCopy._matrixGPU, PnPreCopy._matrixGPU, PnMoyCopy._matrixGPU, PnTildeCopy._matrixGPU, MuLCopy._matrixGPU, nAgentByBusCopy._matrixGPU, _rhol, Ap2Copy._matrixGPU, CpCopy._matrixGPU,
								PminCopy._matrixGPU, PmaxCopy._matrixGPU, Apt1Copy._matrixGPU, Apt2Copy._matrixGPU, Bpt2Copy._matrixGPU, CoresSoloBusAgentCopy._matrixGPU, CoresAgentBusCopy._matrixGPU, CoresAgentBusBeginCopy._matrixGPU, epsL, nIterL, Agent, B);
							break;
						case 16:
							updatePnPGPUSharedResidualTest< 16> << <numBlocks, blockSize >> > (PnCopy._matrixGPU, PnPreCopy._matrixGPU, PnMoyCopy._matrixGPU, PnTildeCopy._matrixGPU, MuLCopy._matrixGPU, nAgentByBusCopy._matrixGPU, _rhol, Ap2Copy._matrixGPU, CpCopy._matrixGPU,
								PminCopy._matrixGPU, PmaxCopy._matrixGPU, Apt1Copy._matrixGPU, Apt2Copy._matrixGPU, Bpt2Copy._matrixGPU, CoresSoloBusAgentCopy._matrixGPU, CoresAgentBusCopy._matrixGPU, CoresAgentBusBeginCopy._matrixGPU, epsL, nIterL, Agent, B);
							break;
						case  8:
							updatePnPGPUSharedResidualTest<  8> << <numBlocks, blockSize >> > (PnCopy._matrixGPU, PnPreCopy._matrixGPU, PnMoyCopy._matrixGPU, PnTildeCopy._matrixGPU, MuLCopy._matrixGPU, nAgentByBusCopy._matrixGPU, _rhol, Ap2Copy._matrixGPU, CpCopy._matrixGPU,
								PminCopy._matrixGPU, PmaxCopy._matrixGPU, Apt1Copy._matrixGPU, Apt2Copy._matrixGPU, Bpt2Copy._matrixGPU, CoresSoloBusAgentCopy._matrixGPU, CoresAgentBusCopy._matrixGPU, CoresAgentBusBeginCopy._matrixGPU, epsL, nIterL, Agent, B);
							break;
						case  4:
							updatePnPGPUSharedResidualTest<  4> << <numBlocks, blockSize >> > (PnCopy._matrixGPU, PnPreCopy._matrixGPU, PnMoyCopy._matrixGPU, PnTildeCopy._matrixGPU, MuLCopy._matrixGPU, nAgentByBusCopy._matrixGPU, _rhol, Ap2Copy._matrixGPU, CpCopy._matrixGPU,
								PminCopy._matrixGPU, PmaxCopy._matrixGPU, Apt1Copy._matrixGPU, Apt2Copy._matrixGPU, Bpt2Copy._matrixGPU, CoresSoloBusAgentCopy._matrixGPU, CoresAgentBusCopy._matrixGPU, CoresAgentBusBeginCopy._matrixGPU, epsL, nIterL, Agent, B);
							break;
						case  2:
							updatePnPGPUSharedResidualTest<  2> << <numBlocks, blockSize >> > (PnCopy._matrixGPU, PnPreCopy._matrixGPU, PnMoyCopy._matrixGPU, PnTildeCopy._matrixGPU, MuLCopy._matrixGPU, nAgentByBusCopy._matrixGPU, _rhol, Ap2Copy._matrixGPU, CpCopy._matrixGPU,
								PminCopy._matrixGPU, PmaxCopy._matrixGPU, Apt1Copy._matrixGPU, Apt2Copy._matrixGPU, Bpt2Copy._matrixGPU, CoresSoloBusAgentCopy._matrixGPU, CoresAgentBusCopy._matrixGPU, CoresAgentBusBeginCopy._matrixGPU, epsL, nIterL, Agent, B);
							break;
						case  1:
							updatePnPGPUSharedResidualTest<  1> << <numBlocks, blockSize >> > (PnCopy._matrixGPU, PnPreCopy._matrixGPU, PnMoyCopy._matrixGPU, PnTildeCopy._matrixGPU, MuLCopy._matrixGPU, nAgentByBusCopy._matrixGPU, _rhol, Ap2Copy._matrixGPU, CpCopy._matrixGPU,
								PminCopy._matrixGPU, PmaxCopy._matrixGPU, Apt1Copy._matrixGPU, Apt2Copy._matrixGPU, Bpt2Copy._matrixGPU, CoresSoloBusAgentCopy._matrixGPU, CoresAgentBusCopy._matrixGPU, CoresAgentBusBeginCopy._matrixGPU, epsL, nIterL, Agent, B);
							break;
						}


						hipDeviceSynchronize();
						b = std::chrono::high_resolution_clock::now();
						break;
					case 1:
						a = std::chrono::high_resolution_clock::now();
						switch (blockSize) {
						case 512:
							updatePnPGPUSharedResidualSameThreadTest<512> << <numBlocks, blockSize >> > (PnCopy._matrixGPU, PnPreCopy._matrixGPU, PnMoyCopy._matrixGPU, PnTildeCopy._matrixGPU, MuLCopy._matrixGPU, nAgentByBusCopy._matrixGPU, _rhol, Ap2Copy._matrixGPU, CpCopy._matrixGPU,
								PminCopy._matrixGPU, PmaxCopy._matrixGPU, Apt1Copy._matrixGPU, Apt2Copy._matrixGPU, Bpt2Copy._matrixGPU, CoresSoloBusAgentCopy._matrixGPU, CoresAgentBusCopy._matrixGPU, CoresAgentBusBeginCopy._matrixGPU, epsL, nIterL, Agent, B);
							break;
						case 256:
							updatePnPGPUSharedResidualSameThreadTest<256> << <numBlocks, blockSize >> > (PnCopy._matrixGPU, PnPreCopy._matrixGPU, PnMoyCopy._matrixGPU, PnTildeCopy._matrixGPU, MuLCopy._matrixGPU, nAgentByBusCopy._matrixGPU, _rhol, Ap2Copy._matrixGPU, CpCopy._matrixGPU,
								PminCopy._matrixGPU, PmaxCopy._matrixGPU, Apt1Copy._matrixGPU, Apt2Copy._matrixGPU, Bpt2Copy._matrixGPU, CoresSoloBusAgentCopy._matrixGPU, CoresAgentBusCopy._matrixGPU, CoresAgentBusBeginCopy._matrixGPU, epsL, nIterL, Agent, B);							break;
						case 128:
							updatePnPGPUSharedResidualSameThreadTest<128> << <numBlocks, blockSize >> > (PnCopy._matrixGPU, PnPreCopy._matrixGPU, PnMoyCopy._matrixGPU, PnTildeCopy._matrixGPU, MuLCopy._matrixGPU, nAgentByBusCopy._matrixGPU, _rhol, Ap2Copy._matrixGPU, CpCopy._matrixGPU,
								PminCopy._matrixGPU, PmaxCopy._matrixGPU, Apt1Copy._matrixGPU, Apt2Copy._matrixGPU, Bpt2Copy._matrixGPU, CoresSoloBusAgentCopy._matrixGPU, CoresAgentBusCopy._matrixGPU, CoresAgentBusBeginCopy._matrixGPU, epsL, nIterL, Agent, B);						break;
						case 64:
							updatePnPGPUSharedResidualSameThreadTest< 64> << <numBlocks, blockSize >> > (PnCopy._matrixGPU, PnPreCopy._matrixGPU, PnMoyCopy._matrixGPU, PnTildeCopy._matrixGPU, MuLCopy._matrixGPU, nAgentByBusCopy._matrixGPU, _rhol, Ap2Copy._matrixGPU, CpCopy._matrixGPU,
								PminCopy._matrixGPU, PmaxCopy._matrixGPU, Apt1Copy._matrixGPU, Apt2Copy._matrixGPU, Bpt2Copy._matrixGPU, CoresSoloBusAgentCopy._matrixGPU, CoresAgentBusCopy._matrixGPU, CoresAgentBusBeginCopy._matrixGPU, epsL, nIterL, Agent, B);
							break;
						case 32:
							updatePnPGPUSharedResidualSameThreadTest< 32> << <numBlocks, blockSize >> > (PnCopy._matrixGPU, PnPreCopy._matrixGPU, PnMoyCopy._matrixGPU, PnTildeCopy._matrixGPU, MuLCopy._matrixGPU, nAgentByBusCopy._matrixGPU, _rhol, Ap2Copy._matrixGPU, CpCopy._matrixGPU,
								PminCopy._matrixGPU, PmaxCopy._matrixGPU, Apt1Copy._matrixGPU, Apt2Copy._matrixGPU, Bpt2Copy._matrixGPU, CoresSoloBusAgentCopy._matrixGPU, CoresAgentBusCopy._matrixGPU, CoresAgentBusBeginCopy._matrixGPU, epsL, nIterL, Agent, B);
							break;
						case 16:
							updatePnPGPUSharedResidualSameThreadTest< 16> << <numBlocks, blockSize >> > (PnCopy._matrixGPU, PnPreCopy._matrixGPU, PnMoyCopy._matrixGPU, PnTildeCopy._matrixGPU, MuLCopy._matrixGPU, nAgentByBusCopy._matrixGPU, _rhol, Ap2Copy._matrixGPU, CpCopy._matrixGPU,
								PminCopy._matrixGPU, PmaxCopy._matrixGPU, Apt1Copy._matrixGPU, Apt2Copy._matrixGPU, Bpt2Copy._matrixGPU, CoresSoloBusAgentCopy._matrixGPU, CoresAgentBusCopy._matrixGPU, CoresAgentBusBeginCopy._matrixGPU, epsL, nIterL, Agent, B);
							break;
						case  8:
							updatePnPGPUSharedResidualSameThreadTest<  8> << <numBlocks, blockSize >> > (PnCopy._matrixGPU, PnPreCopy._matrixGPU, PnMoyCopy._matrixGPU, PnTildeCopy._matrixGPU, MuLCopy._matrixGPU, nAgentByBusCopy._matrixGPU, _rhol, Ap2Copy._matrixGPU, CpCopy._matrixGPU,
								PminCopy._matrixGPU, PmaxCopy._matrixGPU, Apt1Copy._matrixGPU, Apt2Copy._matrixGPU, Bpt2Copy._matrixGPU, CoresSoloBusAgentCopy._matrixGPU, CoresAgentBusCopy._matrixGPU, CoresAgentBusBeginCopy._matrixGPU, epsL, nIterL, Agent, B);
							break;
						case  4:
							updatePnPGPUSharedResidualSameThreadTest<  4> << <numBlocks, blockSize >> > (PnCopy._matrixGPU, PnPreCopy._matrixGPU, PnMoyCopy._matrixGPU, PnTildeCopy._matrixGPU, MuLCopy._matrixGPU, nAgentByBusCopy._matrixGPU, _rhol, Ap2Copy._matrixGPU, CpCopy._matrixGPU,
								PminCopy._matrixGPU, PmaxCopy._matrixGPU, Apt1Copy._matrixGPU, Apt2Copy._matrixGPU, Bpt2Copy._matrixGPU, CoresSoloBusAgentCopy._matrixGPU, CoresAgentBusCopy._matrixGPU, CoresAgentBusBeginCopy._matrixGPU, epsL, nIterL, Agent, B);
							break;
						case  2:
							updatePnPGPUSharedResidualSameThreadTest<  2> << <numBlocks, blockSize >> > (PnCopy._matrixGPU, PnPreCopy._matrixGPU, PnMoyCopy._matrixGPU, PnTildeCopy._matrixGPU, MuLCopy._matrixGPU, nAgentByBusCopy._matrixGPU, _rhol, Ap2Copy._matrixGPU, CpCopy._matrixGPU,
								PminCopy._matrixGPU, PmaxCopy._matrixGPU, Apt1Copy._matrixGPU, Apt2Copy._matrixGPU, Bpt2Copy._matrixGPU, CoresSoloBusAgentCopy._matrixGPU, CoresAgentBusCopy._matrixGPU, CoresAgentBusBeginCopy._matrixGPU, epsL, nIterL, Agent, B);
							break;
						case  1:
							updatePnPGPUSharedResidualSameThreadTest<  1> << <numBlocks, blockSize >> > (PnCopy._matrixGPU, PnPreCopy._matrixGPU, PnMoyCopy._matrixGPU, PnTildeCopy._matrixGPU, MuLCopy._matrixGPU, nAgentByBusCopy._matrixGPU, _rhol, Ap2Copy._matrixGPU, CpCopy._matrixGPU,
								PminCopy._matrixGPU, PmaxCopy._matrixGPU, Apt1Copy._matrixGPU, Apt2Copy._matrixGPU, Bpt2Copy._matrixGPU, CoresSoloBusAgentCopy._matrixGPU, CoresAgentBusCopy._matrixGPU, CoresAgentBusBeginCopy._matrixGPU, epsL, nIterL, Agent, B);
							break;
						}
						hipDeviceSynchronize();
						b = std::chrono::high_resolution_clock::now();
						break;
					default:
						return 0;
						break;
					}
					time += std::chrono::duration_cast<std::chrono::nanoseconds>(b - a).count();
				}
				temps.set(i * nNAgent + j, simu, (float)time / nRepet);
			}
		}		
	}
	nBusMat.saveCSV(fileName, mode);
	nAgentMat.saveCSV(fileName, mode);
	temps.saveCSV(fileName, mode);
	return temps.sum();
}



template <unsigned int blockSize>
__global__ void updatePnPGPUSharedResidualTest(float* Pn, float* PnPre, float* PnMoy, float* PnTilde, float* MUL, float* nAgentByBus, float _rhol, float* Ap2, float* Cp, float* Pmin,
	float* Pmax, float* Apt1, float* Apt2, float* Bpt2, float* CoresSoloBusAgent, float* CoresBusAgent, float* CoresBusAgentBegin, float eps, int nIterLMax, int nAgent, int nBus) {

	//Definition de toutes les variables locales
	int i = blockIdx.x; // c'est aussi l'identifiant du bus !
	unsigned int thIdx = threadIdx.x;

	// ne change pas

	float Ap2local;
	float Ap12local;
	float Cplocal;
	float Pminlocal;
	float Pmaxlocal;
	float Pnlocal; // change
	float Pnprelocal; // change

	float bpt, MULOCAL, moy, p;
	float m, r, ub, lb, t;
	// le changement doit �tre partag� par tous les threads du bloc

	__shared__ float MuShared[2];
	__shared__ float PnMoyShared[2];
	__shared__ float PnTildeShared[2];
	__shared__ bool mustContinue;

	// constant et commun � tous les thread d'un bloc
	__shared__ float Apt1Shared;
	__shared__ float Apt2Shared;
	__shared__ float Apt12Shared;
	__shared__ float Bpt2Shared[2];
	__shared__ int nAgentShared;
	__shared__ float at1Shared;

	__shared__ float shArrP[blockSize / 2 + 1];
	__shared__ float shArrQ[blockSize / 2 + 1];

	if (thIdx < (blockSize / 2 + 1)) {
		shArrP[thIdx] = 0;
		shArrQ[thIdx] = 0;
	}
	if (thIdx < 1) {
		Apt1Shared = Apt1[i]; // rho_l *Ni, m�me pour les 2
		Apt2Shared = Apt2[i]; // rho * Ni^2, m�me pour les 2
		Apt12Shared = Apt1Shared + Apt2Shared; // m�me pour les 2
		nAgentShared = nAgentByBus[i];
		at1Shared = _rhol;
		mustContinue = false;
	}


	if (thIdx < 2) {
		Bpt2Shared[thIdx] = Bpt2[i + nBus * thIdx];
		MuShared[thIdx] = MUL[i + nBus * thIdx];
		PnMoyShared[thIdx] = PnMoy[i + nBus * thIdx];
		PnTildeShared[thIdx] = PnTilde[i + nBus * thIdx];
	}
	__syncthreads();

	int iter = 0;
	if (nAgentShared > 0) { // sinon il n'y a rien � faire
		int indicePorQ = thIdx / nAgentShared; // 0 or 1
		const int CoresAgentLinLocal = CoresBusAgentBegin[i];
		const int j = CoresAgentLinLocal + thIdx;
		double res = 0;
		if (nAgentShared == 1) { // cas trivial s'il n'y a qu'un agent, la divergence est entre les blocs donc c'est ok
			if (thIdx < 2) {
				int agent = CoresSoloBusAgent[i];
				// Cplocal et Ap12local, Pmaxlocal, Pminlocal � definir
				Cplocal = Cp[agent + thIdx * nAgent];
				Ap2local = Ap2[agent + thIdx * nAgent];
				ub = Pmax[agent + thIdx * nAgent];
				lb = Pmin[agent + thIdx * nAgent];
				r = (Apt2Shared * Bpt2Shared[thIdx] - Cplocal) / (Apt2Shared + Ap2local); //pn = (_rho * Bpt2.get(b, 0) - Cost2.get(n, 0)) / ((_rho + Cost1.get(n, 0)));
				t = (ub - r) * (r > ub) + (lb - r) * (r < lb) + r;
				Pnlocal = t;
				Pnprelocal = t;
				PnMoyShared[thIdx] = t;
				PnTildeShared[thIdx] = t;// PnMoyShared[thIdx];
			}
		}
		else {
			Pnlocal = 0;
			if (thIdx < 2* nAgentShared)
			{
				int indice = j < nAgentShared ? j : j - nAgentShared;
				int agent = CoresBusAgent[indice] + indicePorQ * nAgent;
				// P & Q
				Ap2local = Ap2[agent];
				Ap12local = Ap2local + _rhol;
				Cplocal = Cp[agent];
				Pminlocal = Pmin[agent];
				Pmaxlocal = Pmax[agent];
				Pnlocal = Pn[agent];
			}

			//Calcul des it�rations

			for (iter = 0; iter < nIterLMax; iter++) {
				__syncthreads();
				if (thIdx < 2 * nAgentShared) {
					MULOCAL = MuShared[indicePorQ]; // il y a 2 valeurs c'est tr�s chiant, ce n'est pas broadcast
					moy = PnMoyShared[indicePorQ]; // avec de la chance cela fait 2 broadcasts
					p = PnTildeShared[indicePorQ];
					// P & Q
					Pnprelocal = Pnlocal;
					m = Pnlocal - moy + p - MULOCAL; // Pn.get(i, 0) - PnMoy.get(bus, 0) + PnTilde.get(bus, 0) - MuL.get(bus, 0);
					r = (m * at1Shared - Cplocal) / Ap12local; // pn = (Bp1.get(n, 0) * _rhol - Cost2.get(n, 0)) / Ap12.get(n, 0);
					ub = Pmaxlocal;
					lb = Pminlocal;
					t = (ub - r) * (r > ub) + (lb - r) * (r < lb) + r;
					Pnlocal = t;
					res = (double)t - Pnprelocal;
					res = res * res;
					if (res > eps) {
						mustContinue = true; // pas de race condition, car l'ordre n'importe pas,
					}
				}
				if (thIdx < nAgentShared) {
					shArrP[thIdx] = Pnlocal;
				}
				else if (thIdx < 2*nAgentShared)
				{
					shArrQ[thIdx - nAgentShared] = Pnlocal;
				}

				__syncthreads();
				if (blockSize >= 512) {
					if (thIdx < 128) {
						shArrP[thIdx] += shArrP[thIdx + 128];
					}
					else if (thIdx < 256)
					{
						shArrQ[thIdx - 128] += shArrQ[thIdx];
					}
					__syncthreads();
				}
				if (blockSize >= 256) {
					if (thIdx < 64) {
						shArrP[thIdx] += shArrP[thIdx + 64];
					}
					else if (thIdx < 128)
					{
						shArrQ[thIdx - 64] += shArrQ[thIdx];
					}
					__syncthreads();
				}
				if (blockSize >= 128) {
					if (thIdx < 32) {
						warpReduce<blockSize>(shArrP, thIdx);
					}
					else if (thIdx < 64)
					{
						warpReduce<blockSize>(shArrQ, thIdx - 32);
					}

				}
				__syncthreads();

				if (thIdx < 2) {
					moy = (shArrP[0] + thIdx * (shArrQ[0] - shArrP[0])) / nAgentShared;
					PnMoyShared[thIdx] = moy;
					bpt = moy + MuShared[thIdx]; //Bpt1.set(b, 0, MuL.get(b, 0) + PnMoy.get(b, 0));
					p = (Apt1Shared * bpt + Apt2Shared * Bpt2Shared[thIdx]) / Apt12Shared; //pn = (Bpt1.get(b, 0) * Apt1.get(b, 0) + Bpt2.get(b, 0) * Apt2.get(b, 0)) / Apt12.get(b, 0);
					PnTildeShared[thIdx] = p;
					res = p - moy;
					res = res * res;
					if (res > eps) {
						mustContinue = true;
					}
					MuShared[thIdx] = MuShared[thIdx] + moy - p; // mu = MuL.get(b, 0) + PnMoy.get(b, 0) - PnTilde.get(b, 0);
				}
				__syncthreads();
				if (!mustContinue) {
					break;
				}
				else {
					__syncthreads();
					if (thIdx == 0) {
						mustContinue = false;
					}
				}
			}
		}
		//Ecriture des it�rations
		__syncthreads();

		if (thIdx < 2*nAgentShared)
		{
			int indice = j - indicePorQ * nAgentShared;
			int agent = CoresBusAgent[indice] + indicePorQ * nAgent;

			Pn[agent] = Pnlocal;
			PnPre[agent] = Pnprelocal;

		}
		if (thIdx < 2) {
			PnMoy[blockIdx.x + thIdx * nBus] = PnMoyShared[thIdx];// TMoyShared;
			PnTilde[blockIdx.x + thIdx * nBus] = PnTildeShared[thIdx];// PShared;
			MUL[blockIdx.x + thIdx * nBus] = MuShared[thIdx];// MuShared;
		}
	}
}



template <unsigned int blockSize>
__global__ void updatePnPGPUSharedResidualSameThreadTest(float* Pn, float* PnPre, float* PnMoy, float* PnTilde, float* MUL, float* nAgentByBus, float _rhol, float* Ap2, float* Cp, float* Pmin,
	float* Pmax, float* Apt1, float* Apt2, float* Bpt2, float* CoresSoloBusAgent, float* CoresBusAgent, float* CoresBusAgentBegin, float eps, int nIterLMax, int nAgent, int nBus) {

	//Definition de toutes les variables locales
	int i = blockIdx.x; // c'est aussi l'identifiant du bus !
	unsigned int thIdx = threadIdx.x;

	// ne change pas

	float Ap2local[2];
	float Ap12local[2];
	float Cplocal[2];
	float Pminlocal[2];
	float Pmaxlocal[2];
	float Pnlocal[2]; // change
	float Pnprelocal[2]; // change

	float bpt, MULOCAL, moy, p;
	float m, r, ub, lb, t;
	// le changement doit �tre partag� par tous les threads du bloc

	__shared__ float MuShared[2];
	__shared__ float PnMoyShared[2];
	__shared__ float PnTildeShared[2];
	__shared__ bool mustContinue;

	// constant et commun � tous les thread d'un bloc
	__shared__ float Apt1Shared;
	__shared__ float Apt2Shared;
	__shared__ float Apt12Shared;
	__shared__ float Bpt2Shared[2];
	__shared__ int nAgentShared;
	__shared__ float at1Shared;

	__shared__ float shArrP[blockSize / 2 + 1];
	__shared__ float shArrQ[blockSize / 2 + 1];

	if (thIdx < (blockSize / 2 + 1)) {
		shArrP[thIdx] = 0;
		shArrQ[thIdx] = 0;
	}
	if (thIdx < 1) {
		Apt1Shared = Apt1[i]; // rho_l *Ni, m�me pour les 2
		Apt2Shared = Apt2[i]; // rho * Ni^2, m�me pour les 2
		Apt12Shared = Apt1Shared + Apt2Shared; // m�me pour les 2
		nAgentShared = nAgentByBus[i];
		at1Shared = _rhol;
		mustContinue = false;
	}


	if (thIdx < 2) {
		Bpt2Shared[thIdx] = Bpt2[i + nBus * thIdx];
		MuShared[thIdx] = MUL[i + nBus * thIdx];
		PnMoyShared[thIdx] = PnMoy[i + nBus * thIdx];
		PnTildeShared[thIdx] = PnTilde[i + nBus * thIdx];
	}
	__syncthreads();

	int iter = 0;
	if (nAgentShared > 0) { // sinon il n'y a rien � faire
		const int CoresAgentLinLocal = CoresBusAgentBegin[i];
		const int j = CoresAgentLinLocal + thIdx;
		double res = 0;
		if (nAgentShared == 1) { // cas trivial s'il n'y a qu'un agent, la divergence est entre les blocs donc c'est ok
			if (thIdx == 0) {
				int agent = CoresSoloBusAgent[i];
				// Cplocal et Ap12local, Pmaxlocal, Pminlocal � definir
				Cplocal[0] = Cp[agent];
				Ap2local[0] = Ap2[agent];
				ub = Pmax[agent];
				lb = Pmin[agent];
				r = (Apt2Shared * Bpt2Shared[0] - Cplocal[0]) / (Apt2Shared + Ap2local[0]); //pn = (_rho * Bpt2.get(b, 0) - Cost2.get(n, 0)) / ((_rho + Cost1.get(n, 0)));
				t = (ub - r) * (r > ub) + (lb - r) * (r < lb) + r;
				Pnlocal[0] = t;
				Pnprelocal[0] = t;
				PnMoyShared[0] = t;
				PnTildeShared[0] = t;// PnMoyShared[thIdx];

				// Q 
				Cplocal[1] = Cp[agent + nAgent];
				Ap2local[1] = Ap2[agent + nAgent];
				ub = Pmax[agent + nAgent];
				lb = Pmin[agent + nAgent];
				r = (Apt2Shared * Bpt2Shared[1] - Cplocal[1]) / (Apt2Shared + Ap2local[1]); //pn = (_rho * Bpt2.get(b, 0) - Cost2.get(n, 0)) / ((_rho + Cost1.get(n, 0)));
				t = (ub - r) * (r > ub) + (lb - r) * (r < lb) + r;
				Pnlocal[1] = t;
				Pnprelocal[1] = t;
				PnMoyShared[1] = t;
				PnTildeShared[1] = t;// PnMoyShared[thIdx];
			}
		}
		else {
			Pnlocal[0] = 0;
			Pnlocal[1] = 0;
			if (thIdx < nAgentShared)
			{
				int agent = CoresBusAgent[j];
				// P & Q
				Ap2local[0] = Ap2[agent];
				Ap12local[0] = Ap2local[0] + _rhol;
				Cplocal[0] = Cp[agent];
				Pminlocal[0] = Pmin[agent];
				Pmaxlocal[0] = Pmax[agent];
				Pnlocal[0] = Pn[agent];

				Ap2local[1] = Ap2[agent + nAgent];
				Ap12local[1] = Ap2local[1] + _rhol;
				Cplocal[1] = Cp[agent + nAgent];
				Pminlocal[1] = Pmin[agent + nAgent];
				Pmaxlocal[1] = Pmax[agent + nAgent];
				Pnlocal[1] = Pn[agent + nAgent];
			}

			//Calcul des it�rations

			for (iter = 0; iter < nIterLMax; iter++) {
				__syncthreads();
				if (thIdx < nAgentShared) {
					// P
					MULOCAL = MuShared[0]; 
					moy = PnMoyShared[0]; 
					p = PnTildeShared[0];
					
					Pnprelocal[0] = Pnlocal[0];
					m = Pnlocal[0] - moy + p - MULOCAL; // Pn.get(i, 0) - PnMoy.get(bus, 0) + PnTilde.get(bus, 0) - MuL.get(bus, 0);
					r = (m * at1Shared - Cplocal[0]) / Ap12local[0]; // pn = (Bp1.get(n, 0) * _rhol - Cost2.get(n, 0)) / Ap12.get(n, 0);
					ub = Pmaxlocal[0];
					lb = Pminlocal[0];
					t = (ub - r) * (r > ub) + (lb - r) * (r < lb) + r;
					Pnlocal[0] = t;
					res = (double)t - Pnprelocal[0];
					res = res * res;
					if (res > eps) {
						mustContinue = true; // pas de race condition, car l'ordre n'importe pas,
					}
					// Q
					MULOCAL = MuShared[1];
					moy = PnMoyShared[1];
					p = PnTildeShared[1];

					Pnprelocal[1] = Pnlocal[1];
					m = Pnlocal[1] - moy + p - MULOCAL; // Pn.get(i, 0) - PnMoy.get(bus, 0) + PnTilde.get(bus, 0) - MuL.get(bus, 0);
					r = (m * at1Shared - Cplocal[1]) / Ap12local[1]; // pn = (Bp1.get(n, 0) * _rhol - Cost2.get(n, 0)) / Ap12.get(n, 0);
					ub = Pmaxlocal[1];
					lb = Pminlocal[1];
					t = (ub - r) * (r > ub) + (lb - r) * (r < lb) + r;
					Pnlocal[1] = t;
					res = (double)t - Pnprelocal[1];
					res = res * res;
					if (res > eps) {
						mustContinue = true; // pas de race condition, car l'ordre n'importe pas,
					}

				}
				
				shArrP[thIdx] = Pnlocal[0];
				shArrQ[thIdx] = Pnlocal[1];
				
				
				__syncthreads();
				if (blockSize >= 512) {
					if (thIdx < 256) {
						shArrP[thIdx] += shArrP[thIdx + 256];
						shArrQ[thIdx] += shArrQ[thIdx + 256];
					}
					__syncthreads();
				}
				if (blockSize >= 256) {
					if (thIdx < 128) {
						shArrP[thIdx] += shArrP[thIdx + 128];
						shArrQ[thIdx] += shArrQ[thIdx + 128];
					}
					__syncthreads();
				}
				if (blockSize >= 128) {
					if (thIdx < 64) {
						shArrP[thIdx] += shArrP[thIdx + 64];
						shArrQ[thIdx] += shArrQ[thIdx + 64];
					}
					__syncthreads();
				}
				if (blockSize >= 64) {
					if (thIdx < 32) {
						warpReduce<blockSize>(shArrP, thIdx);
						warpReduce<blockSize>(shArrQ, thIdx);
					}
				}
				__syncthreads();

				if (thIdx == 0) {
					// P
					moy = shArrP[0] / nAgentShared;
					PnMoyShared[0] = moy;
					bpt = moy + MuShared[0]; //Bpt1.set(b, 0, MuL.get(b, 0) + PnMoy.get(b, 0));
					p = (Apt1Shared * bpt + Apt2Shared * Bpt2Shared[0]) / Apt12Shared; //pn = (Bpt1.get(b, 0) * Apt1.get(b, 0) + Bpt2.get(b, 0) * Apt2.get(b, 0)) / Apt12.get(b, 0);
					PnTildeShared[0] = p;
					res = p - moy;
					res = res * res;
					if (res > eps) {
						mustContinue = true;
					}
					MuShared[0] = MuShared[0] + moy - p; // mu = MuL.get(b, 0) + PnMoy.get(b, 0) - PnTilde.get(b, 0);
					// Q
					moy = shArrQ[0] / nAgentShared;
					PnMoyShared[1] = moy;
					bpt = moy + MuShared[1]; //Bpt1.set(b, 0, MuL.get(b, 0) + PnMoy.get(b, 0));
					p = (Apt1Shared * bpt + Apt2Shared * Bpt2Shared[1]) / Apt12Shared; //pn = (Bpt1.get(b, 0) * Apt1.get(b, 0) + Bpt2.get(b, 0) * Apt2.get(b, 0)) / Apt12.get(b, 0);
					PnTildeShared[1] = p;
					res = p - moy;
					res = res * res;
					if (res > eps) {
						mustContinue = true;
					}
					MuShared[1] = MuShared[1] + moy - p; // mu = MuL.get(b, 0) + PnMoy.get(b, 0) - PnTilde.get(b, 0);

				}
				__syncthreads();
				if (!mustContinue) {
					break;
				}
				else {
					__syncthreads();
					if (thIdx == 0) {
						mustContinue = false;
					}
				}
			}
		}
		//Ecriture des it�rations
		__syncthreads();

		if (thIdx < nAgentShared)
		{
			int agent = CoresBusAgent[j];

			Pn[agent] = Pnlocal[0];
			PnPre[agent] = Pnprelocal[0];
			Pn[agent + nAgent] = Pnlocal[1];
			PnPre[agent + nAgent] = Pnprelocal[1];

		}
		if (thIdx == 0) {
			PnMoy[blockIdx.x] = PnMoyShared[0];// TMoyShared;
			PnTilde[blockIdx.x] = PnTildeShared[0];// PShared;
			MUL[blockIdx.x] = MuShared[0];// MuShared;
			PnMoy[blockIdx.x + nBus] = PnMoyShared[1];// TMoyShared;
			PnTilde[blockIdx.x + nBus] = PnTildeShared[1];// PShared;
			MUL[blockIdx.x + nBus] = MuShared[1];// MuShared;
		}
	}
}


// --------------------------------- Voltage GS -------------------------------------------------------


float testCalculVGS(int method) {
	std::string fileName = "TempsVGS" + std::to_string(method) + ".csv";
	//float elapsedTime;
	std::chrono::high_resolution_clock::time_point a;
	std::chrono::high_resolution_clock::time_point b;
	unsigned int time;
	std::ios_base::openmode mode = std::fstream::in | std::fstream::out | std::fstream::app;
	const int nNBus = 3;
	const int nNLine = 5;
	const int nSimu = 100;
	const int nRepet = 10;
	int nBus[nNBus] = { 10, 100, 500};
	// nombre ligne : radial, 1/2 milieu,  milieu , 3/2 milieu  full connecté
	int nLineMax = 70000; // pouréviter les badalloc
	int blockSize = 256;
	float values[nSimu];
	float values2[nSimu];

	MatrixCPU temps(nNBus * nNLine, nSimu, 0);
	MatrixCPU nBusMat(1, nNBus);
	MatrixCPU nLineMat(1, nNLine * nNBus);
	for (int j = 0; j < nSimu; j++) {
		values[j] = (float)(rand()) / rand();
		values2[j] = (float)(rand()) / rand();
	}
	int indice = 0;
	for (int i = 0; i < nNBus; i++) {
		nBusMat.set(0, i, nBus[i]);
		for (int j = 0; j < nNLine; j++) {
			int nLine = 0;
			int nLineFully = nBus[i] * (nBus[i] - 1) / 2;
			nLineFully = MYMIN(nLineFully, nLineMax);
			switch (j)
			{
			case 0:
				nLine = nBus[i] - 1;
				break;
			case 1:
				nLine = 0.25 * nLineFully;
				break;
			case 2:
				nLine =  0.5 * nLineFully;
				break;
			case 3:
				nLine = 0.75 * nLineFully;
				break;
			case 4:
				nLine = nLineFully;
				break;
			default:
				throw std::invalid_argument("wrong number of differents number of line");
			}
			nLineMat.set(0, indice, nLine);
			std::cout << indice << " iteration (" << i << ", " << j << ") nBus " << nBus[i] << " nline " << nLine << std::endl;
			for (int simu = 0; simu < nSimu; simu++) {
				int B = nBus[i];
				MatrixGPU VoltageRealIm(2 * B, 1, values[j] + values2[j], 1);
				MatrixGPU RMGgrid(B + 2 * nLine, 1, values[j], 1);
				MatrixGPU RPGgrid(B + 2 * nLine, 1, values2[j], 1);
				MatrixGPU W0(2 * B, 1, values[j] - values2[j], 1);
				MatrixGPU Rgrid(B + 2 * nLine, 1, values2[j] * 2, 1);
				MatrixGPU Xgrid(B + 2 * nLine, 1, values[j] *2, 1);
				MatrixGPU CoresTrans(B + 2 * nLine, 1, 0, 1);
				

				time = 0;
				int numBlocks;
				for (int repet = 0; repet < nRepet; repet++) {
					StudyCase cas;
					if (j) {
						cas.genGridHTB(B, nLine, 1, 10, 1);
					}
					else {
						cas.genGridBT(B, B, B, 10, 1);
					}
					
					MatrixGPU VoltageRealImCopy(VoltageRealIm);
					MatrixGPU RMGgridCopy(RMGgrid);
					MatrixGPU RPGgridCopy(RPGgrid);
					MatrixGPU W0Copy(W0);
					MatrixGPU RgridCopy(Rgrid);
					MatrixGPU XgridCopy(Xgrid);
			
					MatrixGPU CoresVoiLin(cas.getCoresVoiLin());
					MatrixGPU CoresBusLin(cas.getCoresBusLin());
					MatrixGPU nLines(cas.getNLines());
					
					if (method) {
						CoresTrans.transferCPU();
						int* decompte = new int[B];
						for (int i = 0; i < B; i++) {
							decompte[i] = 0;
						}


						for (int i = 0; i < B; i++) {
							int begin = CoresBusLin.get(i, 0);
							for (int l = begin + 1; l < (begin + nLines.get(i, 0)); l++) { // l = (j, i)
								int j = CoresVoiLin.get(l, 0);
								CoresTrans.set(l, 0, CoresBusLin.get(j, 0) + 1 + decompte[j]);
								decompte[j]++;
							}
						}


						DELETEA(decompte);

						
						CoresTrans.transferGPU();
					}
					CoresVoiLin.transferGPU();
					CoresBusLin.transferGPU();
					nLines.transferGPU();
					

					
					int L = nLine;
					int BL2 = B + 2 * L;
					int B2 = 2 * B;
					hipDeviceSynchronize();
					switch (method)
					{
					case 0:
						numBlocks = 1;
						a = std::chrono::high_resolution_clock::now();
						calculVoltOneStepTest<256> << <1, blockSize, B2 * sizeof(float) >> > (VoltageRealImCopy._matrixGPU, W0Copy._matrixGPU, RgridCopy._matrixGPU, XgridCopy._matrixGPU, RMGgridCopy._matrixGPU, RPGgridCopy._matrixGPU, CoresVoiLin._matrixGPU, CoresBusLin._matrixGPU, nLines._matrixGPU, B);
						hipDeviceSynchronize();
						b = std::chrono::high_resolution_clock::now();
						break;
					case 1:
						numBlocks = B;
						a = std::chrono::high_resolution_clock::now();
						calculVoltStep1Test<256> << <numBlocks, blockSize, B2 * sizeof(float) >> > (VoltageRealImCopy._matrixGPU, W0Copy._matrixGPU, RgridCopy._matrixGPU, XgridCopy._matrixGPU, RMGgridCopy._matrixGPU, RPGgridCopy._matrixGPU, CoresVoiLin._matrixGPU, CoresBusLin._matrixGPU, nLines._matrixGPU, B);
						calculVoltStep2bisTest << <1, blockSize, 2 * (BL2 + B) * sizeof(float) >> > (VoltageRealImCopy._matrixGPU, RMGgridCopy._matrixGPU, RPGgridCopy._matrixGPU, CoresVoiLin._matrixGPU, CoresBusLin._matrixGPU, nLines._matrixGPU, CoresTrans._matrixGPU, B, BL2);

						hipDeviceSynchronize();
						b = std::chrono::high_resolution_clock::now();
						break;
					case 2:
						numBlocks = B;
						a = std::chrono::high_resolution_clock::now();
						calculVoltStep1Test<256> << <numBlocks, blockSize, B2 * sizeof(float) >> > (VoltageRealImCopy._matrixGPU, W0Copy._matrixGPU, RgridCopy._matrixGPU, XgridCopy._matrixGPU, RMGgridCopy._matrixGPU, RPGgridCopy._matrixGPU, CoresVoiLin._matrixGPU, CoresBusLin._matrixGPU, nLines._matrixGPU, B);
						calculVoltStep2Test << <1, blockSize, 2 * (BL2 + B) * sizeof(float) >> > (VoltageRealImCopy._matrixGPU, RMGgridCopy._matrixGPU, RPGgrid._matrixGPU, CoresVoiLin._matrixGPU, CoresBusLin._matrixGPU, nLines._matrixGPU, CoresTrans._matrixGPU, B);

						hipDeviceSynchronize();
						b = std::chrono::high_resolution_clock::now();
						break;
					default:
						return 0;
						break;
					}
					time += std::chrono::duration_cast<std::chrono::nanoseconds>(b - a).count();
				}
				temps.set(indice, simu, (float)time / nRepet);
			}
			indice++;
		}
	}
	nBusMat.saveCSV(fileName, mode);
	nLineMat.saveCSV(fileName, mode);
	temps.saveCSV(fileName, mode);
	return temps.sum();
}

template <unsigned int blockSize>
__global__  void calculVoltOneStepTest(float* VoltageRealIm, float* W0, float* Rgrid, float* Xgrid, float* RMGgrid, float* RPGgrid, float* CoresVoiLin, float* CoresBusLin, float* nLines, int B) {

	int thIdx = threadIdx.x;
	//int i = blockIdx.x; un seul bloc
	int size = blockDim.x;

	extern __shared__ float Voltage[];
	__shared__ float shArr[blockSize];
	__shared__ float shArr2[blockSize];
	for (int k = thIdx; k < 2 * B; k += size) {
		Voltage[k] = VoltageRealIm[k];
	}
	__syncthreads();

	for (int i = 1; i < B; i++) { // calcul de i = iter
		float sum = 0;
		float sum2 = 0;
		int begin = CoresBusLin[i];
		int end = begin + nLines[i];

		for (int l = begin + thIdx + 1; l < end; l += size) {
			int k = CoresVoiLin[l];
			sum  += (RMGgrid[l] * Voltage[k] - RPGgrid[l] * Voltage[k + B]);
			sum2 += (RPGgrid[l] * Voltage[k] + RMGgrid[l] * Voltage[k + B]);
		}
		shArr[thIdx] = sum;
		shArr2[thIdx] = sum2;
		__syncthreads();

		if (blockSize >= 512) {
			if (thIdx < 256) {
				shArr[thIdx] += shArr[thIdx + 256];
				shArr2[thIdx] += shArr2[thIdx + 256];
			}
			__syncthreads();
		}
		if (blockSize >= 256) {
			if (thIdx < 128) {
				shArr[thIdx] += shArr[thIdx + 128];
				shArr2[thIdx] += shArr2[thIdx + 128];
			}
			__syncthreads();
		}
		if (blockSize >= 128) {
			if (thIdx < 64) {
				shArr[thIdx] += shArr[thIdx + 64];
				shArr[thIdx] += shArr2[thIdx + 64];
			} __syncthreads();
		}
		if (thIdx < 32) {
			warpReduce<blockSize>(shArr, thIdx);
			warpReduce<blockSize>(shArr2, thIdx);
		}

		if (thIdx == 0) {
			float vi = Voltage[i];
			float wi = Voltage[i + B];
			float r = Rgrid[i];
			float x = Xgrid[i];
			float W0_local = W0[i];
			float W0B_local = W0[i + B];
			                                                    
			float norm = vi * vi + wi * wi;
			float c = (W0_local * vi + W0B_local * wi) / norm;
			float d = (W0_local * wi - W0B_local * vi) / norm;



			VoltageRealIm[i] = -shArr[0] + c * r - d * x;
			VoltageRealIm[i + B] = -shArr2[0] + d * r + c * x;
		}
		__syncthreads();
	}
	for (int k = thIdx; k < 2 * B; k += size) {
		VoltageRealIm[k] = Voltage[k];
	}
	__syncthreads();


}


__global__ void calculVoltStep2Test(float* VoltageRealIm, float* RMGgrid, float* RPGgrid, float* CoresVoiLin, float* CoresBusLin, float* nLines, float* CoresTrans, int B) {

	/* int thIdx = threadIdx.x;
	 int i = blockIdx.x;
	 int size = blockDim.x;
	 int begin = CoresBusLin[i]; // k = CoresBusLin[iter]; !!!
	 int end = begin + nLines[i]; // k + nLines[iter]
	 // il ne faut pas Ypj/Ypp mais bien Yjp/Yjj, donc il faut savoir quel voisin est p...


	if(i>iter){
		 for (int voisin = thIdx + begin + 1; voisin < end; voisin += size) {
			 int p = CoresVoiLin[voisin];

			 if (p == iter ) { // pour trouver quel indice est p, c'est plutôt nul

				 float db1 = RMGgrid[voisin] * VoltageRealIm[iter] - RPGgrid[voisin] * VoltageRealIm[iter + B];
				 float db2 = RPGgrid[voisin] * VoltageRealIm[iter] + RMGgrid[voisin] * VoltageRealIm[iter + B];

				 VoltageRealIm[i] = VoltageRealIm[i] - db1;
				 VoltageRealIm[i + B] = VoltageRealIm[i + B] - db2;
			 }
		 }
	}*/

	int thIdx = threadIdx.x;
	//int i = blockIdx.x; un seul bloc
	int size = blockDim.x;

	extern __shared__ float Voltage[];

	for (int k = thIdx; k < 2 * B; k += size) {
		Voltage[k] = VoltageRealIm[k];
	}
	__syncthreads();


	for (int iter = 0; iter < B - 1; iter++) {
		int begin = CoresBusLin[iter]; // k = CoresBusLin[iter]; !!!
		int end = begin + nLines[iter]; // k + nLines[iter]
		float ei = Voltage[iter];
		float fi = Voltage[iter + B];

		for (int l = thIdx + begin + 1; l < end; l += size) { // voisin
			int j = CoresVoiLin[l];

			if (j > iter) {
				int lTrans = CoresTrans[l]; // accès pas du tout coalescent !!!

				float ri = RMGgrid[lTrans]; // accès pas du tout coalescent mais c'est sur la mémoire partagé
				float li = RPGgrid[lTrans];

				float db1 = ri * ei - li * fi;
				float db2 = li * ei + ri * fi;

				Voltage[j] = Voltage[j] - db1;
				Voltage[j + B] = Voltage[j + B] - db2;
			}
		}
		__syncthreads();
	}

	for (int k = thIdx; k < 2 * B; k += size) {
		VoltageRealIm[k] = Voltage[k];
	}
	__syncthreads();


}

__global__ void calculVoltStep2bisTest(float* VoltageRealIm, float* RMGgrid, float* RPGgrid, float* CoresVoiLin, float* CoresBusLin, float* nLines, float* CoresTrans, int B, int BL2) {


	int thIdx = threadIdx.x;
	//int i = blockIdx.x; un seul bloc
	int size = blockDim.x;


	extern __shared__ float RI[];

	float* Voltage = &RI[2 * BL2];
	for (int l = thIdx; l < BL2; l += size) { // coalecent
		RI[l] = RMGgrid[l];
		RI[l + BL2] = RPGgrid[l];
	}
	for (int k = thIdx; k < 2 * B; k += size) {
		Voltage[k] = VoltageRealIm[k];
	}
	__syncthreads();


	for (int iter = 0; iter < B - 1; iter++) {
		int begin = CoresBusLin[iter]; // k = CoresBusLin[iter]; !!!
		int end = begin + nLines[iter]; // k + nLines[iter]

		float ei = Voltage[iter];
		float fi = Voltage[iter + B];

		for (int l = thIdx + begin + 1; l < end; l += size) { // voisin
			int j = CoresVoiLin[l];

			if (j > iter) {
				int lTrans = CoresTrans[l];

				float ri = RI[lTrans]; // accès pas du tout coalescent mais c'est sur la mémoire partagé
				float li = RI[lTrans + BL2];


				float db1 = ri * ei - li * fi;
				float db2 = li * ei + ri * fi;

				Voltage[j] = Voltage[j] - db1;
				Voltage[j + B] = Voltage[j + B] - db2;
			}
		}
		__syncthreads();
	}
	for (int k = thIdx; k < 2 * B; k += size) {
		VoltageRealIm[k] = Voltage[k];
	}
	__syncthreads();

}



template <unsigned int blockSize>
__global__ void calculVoltStep1Test(float* VoltageRealIm, float* W0, float* Rgrid, float* Xgrid, float* RMGgrid, float* RPGgrid, float* CoresVoiLin, float* CoresBusLin, float* nLines, int B) {

	__shared__ float shArr[blockSize];
	__shared__ float shArr2[blockSize];
	extern __shared__ float shE[];
	int thIdx = threadIdx.x;
	int i = blockIdx.x;
	int step = blockSize;
	int begin = CoresBusLin[i];
	int end = begin + nLines[i];
	int B2 = 2 * B;

	if (i != 0) {
		for (int n = thIdx; n < B2; n += step)
		{
			shE[n] = VoltageRealIm[n];
		}
		__syncthreads();
		float sum = 0;
		float sum2 = 0;
		for (int l = begin + thIdx + 1; l < end; l += step) {
			int k = CoresVoiLin[l];
			if (k > i) {
				sum -= (RMGgrid[l] * shE[k] - RPGgrid[l] * shE[k + B]);
				sum2 -= (RPGgrid[l] * shE[k] + RMGgrid[l] * shE[k + B]);
			}
		}


		shArr[thIdx] = sum;
		shArr2[thIdx] = sum2;
		__syncthreads();

		if (blockSize >= 512) {
			if (thIdx < 256) {
				shArr[thIdx] += shArr[thIdx + 256];
				shArr2[thIdx] += shArr2[thIdx + 256];
			}
			__syncthreads();
		}
		if (blockSize >= 256) {
			if (thIdx < 128) {
				shArr[thIdx] += shArr[thIdx + 128];
				shArr2[thIdx] += shArr2[thIdx + 128];
			}
			__syncthreads();
		}
		if (blockSize >= 128) {
			if (thIdx < 64) {
				shArr[thIdx] += shArr[thIdx + 64];
				shArr[thIdx] += shArr2[thIdx + 64];
			} __syncthreads();
		}
		if (thIdx < 32) {
			warpReduce<blockSize>(shArr, thIdx);
			warpReduce<blockSize>(shArr2, thIdx);
		}
		if (thIdx == 0) {
			float vi = shE[i];
			float wi = shE[i + B];
			float r = Rgrid[i];
			float x = Xgrid[i];
			float W0_local = W0[i];
			float W0B_local = W0[i + B];

			float norm = vi * vi + wi * wi;
			float c = (W0_local * vi + W0B_local * wi) / norm;
			float d = (W0_local * wi - W0B_local * vi) / norm;



			VoltageRealIm[i] = shArr[0] + c * r - d * x;
			VoltageRealIm[i + B] = shArr2[0] + d * r + c * x;
		}
	}
}
