#include "hip/hip_runtime.h"
#include "../head/TestMatrixGPU.cuh"


int testMGPU()
{
	int n = 1;
	if (!testMGPUConstr1()) return n;
	n++;
    if (!testMGPUConstr2()) return n;
    n++;
    if (!testMGPUConstr3()) return n;
    n++;
    if (!testMGPUSet1()) return n;
    n++;
    if (!testMGPUSet2()) return n;
    n++;
    if (!testMGPUSet3()) return n;
    n++;
    if (!testMGPUSet3()) return n;
    n++;
    if (!testMGPUSetForce()) return n;
    n++;
    if (!testMGPUSetGPU()) return n;
    n++;
    if (!testMGPUSetTrans()) return n;
    std::cout << "*************  10   *****************" << std::endl;
    n++;//10
    if (!testMGPUSetBloc()) return n;
    n++; 
    if (!testMGPUTranferG1()) return n;
    n++; 
    if (!testMGPUTranferG2()) return n;
    n++;
    if (!testMGPUTranferC1()) return n;
    n++;
    if (!testMGPUTranferC2()) return n;
    n++;
    if (!testMGPUConv()) return n;
    n++;
    if (!testMGPUAdd1()) return n;
    n++;
    if (!testMGPUAdd2()) return n;
    n++;
    if (!testMGPUAdd3()) return n;
    n++;
    if (!testMGPUAdd4()) return n;
    std::cout << "*************  20   *****************" << std::endl;
    n++;//20
    if (!testMGPUAdd5()) return n;
    n++; 
    if (!testMGPUAdd6()) return n;
    n++;
    if (!testMGPUAddVect1()) return n;
    n++;
    if (!testMGPUAddVect2()) return n;
    n++;
    if (!testMGPUAddTrans1()) return n;
    n++; 
    if (!testMGPUAddTrans2()) return n;
    n++;
    if (!testMGPUSubstract1()) return n;
    n++;
    if (!testMGPUSubstract2()) return n;
    n++;
    if (!testMGPUSubstract3()) return n;
    n++; 
    if (!testMGPUSubstract4()) return n;
    std::cout << "*************  30   *****************" << std::endl;
    n++;//30 
    if (!testMGPUSubstractVect1()) return n;
    n++; 
    if (!testMGPUSubstractVect2()) return n;
    n++;
    if (!testMGPUSubstractTrans1()) return n;
    n++; 
    if (!testMGPUSubstractTrans2()) return n;
    n++;
    if (!testMGPUMultiply()) return n;
    n++; 
    if (!testMGPUMultiply2()) return n;
    n++;
    if (!testMGPUMultiplyT1()) return n;
    n++;
    if (!testMGPUMultiplyT2()) return n;
    n++;
    if (!testMGPUMultiplyT3()) return n;
    n++;
    if (!testMGPUMultiplyT4()) return n;
    std::cout << "*************  40   *****************" << std::endl;
    n++; //40
    if (!testMGPUMultiplyVect()) return n;
    n++;
    if (!testMGPUMultiplyMat()) return n;
    n++;
    if (!testMGPUMultiplyLinearOp()) return n;
    n++;
    if (!testMGPUDivide1()) return n;
    n++; 
    if (!testMGPUDivide2()) return n;
    n++; 
    if (!testMGPUDivide3()) return n;
    n++; 
    if (!testMGPUDivide4()) return n;
    n++;
    if (!testMGPUDivide5()) return n;
    n++;
    if (!testMGPUmoy1()) return n;
    n++; 
    if (!testMGPUmoy2()) return n;
    std::cout << "*************  50   *****************" << std::endl;
    n++;// 50
    if (!testMGPUmoy3()) return n;
    n++; 
    if (!testMGPUmoy4()) return n;
    n++;
    if (!testMGPUProject1()) return n;
    n++;
    if (!testMGPUProject2()) return n;
    n++;
    if (!testMGPUProjectPos()) return n;
    n++; 
    if (!testMGPUProjectNeg()) return n;
    n++; 
    if (!testMGPUSum1()) return n;
    n++; 
    if (!testMGPUSum2()) return n;
    n++;
    if (!testMGPUSum3()) return n;
    n++; 
    if (!testMGPUSum4()) return n;
    std::cout << "*************  60   *****************" << std::endl;
    n++; // 60
    if (!testMGPUSumPartial()) return n;
    n++;
    if (!testMGPUSwap()) return n;
    n++;
    if (!testMGPUDistance()) return n;
    n++;
    if (!testMGPUDistance2()) return n;
    n++;
    if (!testMGPUMax()) return n;
    n++;
    if (!testMGPUMax2()) return n;
    n++; 
    if (!testMGPUMax3()) return n;
    n++;  
    if (!testMGPUDivideGJ1()) return n;
    n++;
    if (!testMGPUDivideGJ2()) return n;
    n++;
    if (!testMGPUSolveSys()) return n;
    n++;

    return 0;

}

bool testMGPUConstr1(){
    std::cout<< "default constructor"<<std::endl;

    MatrixGPU m;

    return true;

}
bool testMGPUConstr2() {
    std::cout << "param constructor" << std::endl;

    int line = 2;
    int column = line + 1;
    float value = 20;
    MatrixGPU m(line,column,value);

    m.display();

    return true;

}
bool testMGPUConstr3() {
    std::cout << "copy constructor" << std::endl;

    int line = 2;
    int column = line + 1;
    float value = 20;
    MatrixCPU m(line, column, value);
    MatrixGPU m2(m);

    m2.display();

    return true;

}


bool testMGPUSet1()
{
    float value2 = 1;
    float value = 4.5;
    int i = 1;
    int j = 2;
    int n = 3;
    MatrixGPU mnull(n, n,value2);
    mnull.set(i, j, value);
    return  (mnull.get(i, j) == value);
}
bool testMGPUSet2()
{
    int n = 3;
    float value = 4;
    int i = 1;
    int j = n;
    MatrixGPU mnull(n, n);
    try
    {
        mnull.set(i, j, value); 
    }
    catch (std::out_of_range&)
    {
        return true;
    }
    return false;
}
bool testMGPUSet3()
{
    float value = 4.5;
    int n = 3;
    MatrixGPU m1(n, n);
    MatrixGPU m11(n, n);
    MatrixGPU m2(n, n, value);
    MatrixGPU m22(n, n, value);
    m1.set(&m2);
    m11.transferGPU();
    m22.transferGPU();
    

    

    m11.set(&m22);
    m11.transferCPU();
    m22.transferCPU();



    return  ((m1.isEqual(&m2)) && (m11.isEqual(&m22)) && (m1.isEqual(&m22)));
}
bool testMGPUSet4()
{
    float value = 4.5;
    int n = 3;
    MatrixGPU m1(n, n);
    MatrixGPU m2(n-1, n, value);
    MatrixGPU m22(n, n, value);
    m22.transferGPU();
    try
    {
        m1.set(&m2); 
    }
    catch (std::invalid_argument&)
    {
        try
        {
            m1.set(&m22); 
        }
        catch (std::invalid_argument&)
        {
            return true;
        }
        return false;
    }
    return false;
}

bool testMGPUSetForce()
{
    float value2 = 1;
    float value = 4.5;
    int i = 1;
    int j = 2;
    int n = 3;
    MatrixGPU mnull(n, n, value2);
    MatrixGPU mnull2(n, n, value2, true);
    mnull2.set(i, j, value, true);
    mnull.set(i, j, value);

    mnull.display();
    mnull2.display(true);

    return  (mnull.get(i, j) == value) && (mnull2.get(i, j, false) == value);
}

bool testMGPUSetGPU()
{
    float value2 = 1;
    float value = 4.5;
    int i = 1;
    int j = 2;
    int n = 3;
    MatrixGPU m(n, n, value2, 1);
    m.set(i, j, value, true);
    float test = m.get(i, j, false);
    m.transferCPU();
    return  (m.get(i, j) == value) && (test ==value);
}

bool testMGPUSetTrans()
{
    int row = 4;
    int col = 5;
    float value1 = 2;
    float value2 = -1;
    MatrixGPU m(row, col, value1);
    MatrixGPU mTrans1(col, row, value1);
    MatrixGPU mTrans2(col, row, 0);
    MatrixGPU mTrans3(col, row, 0, 1);

    m.set(row - 2, col - 2, value2);
    mTrans1.set(col - 2, row - 2, value2);
    mTrans2.setTrans(&m);
    m.transferGPU();
    mTrans3.setTrans(&m);
    mTrans3.transferCPU();

   


    return (mTrans1.isEqual(&mTrans2) && mTrans2.isEqual(&mTrans3));
}

bool testMGPUSetBloc()
{
    int row = 4;
    int col = 5;
    float value1 = 2;
    float value2 = -1;
    MatrixGPU m1(3*row, 4*col, 0, 1);
    MatrixGPU m11(3 * row, 4 * col);
    MatrixGPU m2(row, col, value1);
    
    int iBegin = 0;
    int iEnd = row;
    int jBegin = 0;
    int jEnd = col;

    int iBegin2 = row + 1;
    int iEnd2 = 2*row + 1;
    int jBegin2 = 2 * col + 1;
    int jEnd2 = 3 * col + 1;

    m11.setBloc(iBegin, iEnd, jBegin, jEnd, &m2);
    m11.setBloc(iBegin2, iEnd2, jBegin2, jEnd2, &m2, value2);

    m2.transferGPU();

    m1.setBloc(iBegin, iEnd, jBegin, jEnd, &m2);
    m1.setBloc(iBegin2, iEnd2, jBegin2, jEnd2, &m2, value2);

    m1.transferCPU();

    /*std::cout << "--------------------------" << std::endl;
    m1.display();
    std::cout << "--------------------" << std::endl;
    m11.display();
    std::cout << "--------------------------" << std::endl;*/


    return m1.isEqual(&m11);
}



bool testMGPUTranferG1() {

    int line = 2;
    int column = line + 1;
    float value = 20;
    MatrixGPU m(line, column, value);
    m.transferGPU();
    m.display();

    return true;
}
bool testMGPUTranferG2() {
    int line = 2;
    int column = line + 1;
    float value = 20;
    MatrixGPU m(line, column, value);
    m.transferGPU();

    try
    {
        m.transferGPU();
    }
    catch (const std::domain_error&)
    {
        return true;
    }
    return false;
}
bool testMGPUTranferC1() {
    int line = 2;
    int column = line + 1;
    float value = 20;
    MatrixGPU m(line, column, value);
    m.transferGPU();
    m.transferCPU();
    m.display();
    return true;

}
bool testMGPUTranferC2() {
    int line = 2;
    int column = line + 1;
    float value = 20;
    MatrixGPU m(line, column, value);
    m.transferGPU();
    m.transferCPU();
    try
    {
        m.transferCPU();
    }
    catch (const std::domain_error&)
    {
        return true;
    }
    return false;
}

bool testMGPUConv()
{
    int line = 2;
    int column = line + 1;
    float value = 20;
    MatrixGPU m(line, column, value);
    MatrixCPU m1(line, column, value);
    MatrixCPU m2;
    m.toMatCPU(m2);
    MatrixCPU m3;
    m.transferGPU();
    m.toMatCPU(m3);
    return (m1.isEqual(&m2) && m1.isEqual(&m3));
}

bool testMGPUAdd1()
{
    int line = 100;
    int column = line + 1;
    float value = 1.5;
    float value2 = 1;
    float value3 = value + value2;

    MatrixGPU m1(line, column, value);
    MatrixGPU m2(line, column, value2);
    MatrixGPU m3(line, column, value3); 
    MatrixGPU temp(line, column);
    temp.add(&m1, &m2);
    if (!temp.isEqual(&m3)) return false;
    m1.transferGPU();
    m2.transferGPU();
    temp.transferGPU();
    temp.add(&m1, &m2);
    temp.transferCPU();


    return temp.isEqual(&m3);

}
bool testMGPUAdd2() {
    int line = 100;
    int column = line + 1;
    float value = 1.5;
    float value2 = 1;
    float value3 = value + value2;
    MatrixGPU m1(line, column, value);
    MatrixGPU m3(line, column, value3);
    MatrixGPU temp(line, column);
    temp.add(&m1, value2);
    if (!temp.isEqual(&m3)) return false;
    m1.transferGPU();
    temp.transferGPU();
    temp.add(&m1, value2);
    temp.transferCPU();
    return temp.isEqual(&m3);
    
}
bool testMGPUAdd3()
{
    int line = 2;
    int column = line + 1;
    float value = 1.5;
    float value2 = 1;
    float value3 = value + value2;

    MatrixGPU m1(line, column, value);
    MatrixGPU m11(line, column, value);
    MatrixGPU m2(line, column, value2);
    MatrixGPU m3(line, column, value3); 
    m1.add(&m2);
    
    if (!m1.isEqual(&m3)) return false;

    m11.transferGPU();
    m2.transferGPU();
    m11.add(&m2);
    m11.transferCPU();
    
    return m11.isEqual(&m3);



}
bool testMGPUAdd4()
{
    int line = 2;
    int column = line + 1;
    float value = 1.5;
    float value2 = -1;
    float value3 = value + value2;
    MatrixGPU m1(line, column, value);
    MatrixGPU m11(line, column, value);
    MatrixGPU m3(line, column, value3);
    m1.add(value2);

    if (!m1.isEqual(&m3)) return false;

    m11.transferGPU();
    m11.add(value2);
    m11.transferCPU();

    return m11.isEqual(&m3);
}
bool testMGPUAdd5()
{
    int line = 100;
    int column = line + 1;
    float value = 1.5;
    float value2 = 1;


    MatrixGPU m1(line, column, value);
    MatrixGPU m2(line, column, value2);
    MatrixGPU m3(column, line);
    MatrixGPU m4(line, column);
    m1.transferGPU();
    m2.transferGPU();
    m3.transferGPU();

    try
    {
        m3.add(&m1, &m2);
    }
    catch (std::invalid_argument&)
    {
        try
        {
            m4.add(&m1, &m2);
        }
        catch (std::invalid_argument&)
        {
            return true;
        }
        return false;
    }
    return false;
}
bool testMGPUAdd6()
{
    int line = 100;
    int column = line + 1;
    float value = 1.5;
    float value2 = 1;


    MatrixGPU m1(line, column, value);
    MatrixGPU m2(column, line, value2);
    MatrixGPU m3(line, column);
    m1.transferGPU();
    m2.transferGPU();
    try
    {
        m1.add(&m2);
    }
    catch (std::invalid_argument&)
    {
        try
        {
            m1.add(&m3);
        }
        catch (std::invalid_argument&)
        {
            return true;
        }
        return false;
    }
    return false;
}

bool testMGPUAddVect1()
{
    int line = 100;
    int column = line + 1;
    float value = 1.5;
    float value2 = 1;
    MatrixGPU m1(line, column, value);
    MatrixGPU m4(column, line, value2);
    MatrixGPU vect(line, 1, value2);
    vect.transferGPU();
    try {
        m1.addVector(&m4);
    }
    catch (std::invalid_argument&) {
        try
        {
            m1.addVector(&vect);
        }
        catch (std::invalid_argument&)
        {
            return true;
        }
        return false;
    }
    return false;

}
bool testMGPUAddVect2()
{
    int line = 2;
    int column = line + 1;
    float value = 1.5;
    float value2 = 1;
    MatrixGPU m1(line, column, value);
    MatrixGPU m11(line, column, value);
    MatrixGPU m2(line, column, value2);
    MatrixGPU vect(1, column, value2 - value);
    MatrixGPU vect1(line, 1, value2 - value);

    m1.transferGPU();
    m11.transferGPU();
    vect.transferGPU();
    vect1.transferGPU();

    m1.addVector(&vect); 
    m1.transferCPU();
    if (!m1.isEqual(&m2)) return false;
    m11.addVector(&vect1); 
    m11.transferCPU();
    if (!m2.isEqual(&m11)) return false;

    return true;
}

bool testMGPUAddTrans1()
{
    int line = 4;
    int column = line + 1;
    int i = 1;
    int j = 2;
    float value = 2;
    float value2 = 1;
    float value3 = value + value2;
    float value4 = 4.5;
    float value5 = value + value4;

    MatrixGPU m1(line, column, value);
    MatrixGPU m2(column, line, value2);
    m2.set(i, j, value4);
    MatrixGPU m3(line, column, value3); 
    m3.set(j, i, value5);
    m1.transferGPU();
    m2.transferGPU();
    m1.addTrans(&m2);
    m1.transferCPU();
    return m1.isEqual(&m3);
}
bool testMGPUAddTrans2() {
    int line = 2;
    int column = line + 1;
    float value = 1.5;
    float value2 = 1;

    MatrixGPU m1(line, column, value);
    MatrixGPU m2(line, column, value2);
    MatrixGPU m3(column, line, value2);
    m3.transferGPU();
    try
    {
        m1.addTrans(&m2);
    }
    catch (std::invalid_argument&)
    {
        try
        {
            m1.addTrans(&m3);
        }
        catch (std::invalid_argument&)
        {
            return true;
        }
        return false;
    }

    return false;
}


bool testMGPUSubstract1()
{
    int line = 2;
    int column = line + 1;
    float value = 1.5;
    float value2 = 1;
    float value3 = value - value2;

    MatrixGPU m1(line, column, value);
    MatrixGPU m2(line, column, value2);
    MatrixGPU m3(line, column, value3); 
    MatrixGPU temp(line, column);
    m1.transferGPU();
    m2.transferGPU();
    temp.transferGPU();
    temp.subtract(&m1, &m2);
    temp.transferCPU();
    return temp.isEqual(&m3);

}
bool testMGPUSubstract2()
{
    int line = 2;
    int column = line + 1;
    float value = 1.5;
    float value2 = 1;
    float value3 = value - value2;

    MatrixGPU m1(line, column, value);
    MatrixGPU m2(line, column, value2);
    MatrixGPU m3(line, column, value3); 
    m1.transferGPU();
    m2.transferGPU();
    m1.subtract(&m2);
    m1.transferCPU();
    return m1.isEqual(&m3);

}
bool testMGPUSubstract3()
{
    int line = 2;
    int column = line + 1;
    float value = 1.5;
    float value2 = 1;

    MatrixGPU m1(line, column, value);
    MatrixGPU m2(line, column, value2);
    MatrixGPU m3(column, line);
    MatrixGPU m4(line, column);
    m4.transferGPU();
    try
    {
        m3.subtract(&m1, &m2);
    }
    catch (std::invalid_argument&)
    {
        try
        {
            m4.subtract(&m1, &m2);
        }
        catch (std::invalid_argument&)
        {
            return true;
        }
        return false;
    }
    return false;
}
bool testMGPUSubstract4()
{
    int line = 100;
    int column = line + 1;
    float value = 1.5;
    float value2 = 1;


    MatrixGPU m1(line, column, value);
    MatrixGPU m2(column, line, value2);
    MatrixGPU m3(line, column);
    m1.transferGPU();
    m2.transferGPU();
    try
    {
        m1.subtract(&m2);
    }
    catch (std::invalid_argument&)
    {
        try
        {
            m1.subtract(&m3);
        }
        catch (std::invalid_argument&)
        {
            return true;
        }
        return false;
    }
    return false;
}

bool testMGPUSubstractVect1()
{
    int line = 2;
    int column = line + 1;
    float value = 1.5;
    float value2 = 1;
    MatrixGPU m1(line, column, value);
    MatrixGPU m4(column, line, value2);
    MatrixGPU vect(1, line, value2);

    try {
        m1.subtractVector(&m4);
    }
    catch (std::invalid_argument&) {
        try
        {
            m1.subtractVector(&vect);
        }
        catch (std::invalid_argument&)
        {
            return true;
        }
        return false;
    }
    return false;

}
bool testMGPUSubstractVect2()
{
    int line = 2;
    int column = line + 1;
    float value = 1.5;
    float value2 = 1;
    MatrixGPU m1(line, column, value);
    MatrixGPU m11(line, column, value);
    MatrixGPU m2(line, column, value2);
    MatrixGPU vect(1, column, value - value2);
    MatrixGPU vect1(line, 1, value - value2);

    m1.transferGPU();
    vect.transferGPU();
    vect1.transferGPU();
    m11.transferGPU();
    m1.subtractVector(&vect);
    m1.transferCPU();
    if (!m1.isEqual(&m2)) return false;

    m11.subtractVector(&vect1);
    m11.transferCPU();
    if (!m2.isEqual(&m11)) return false;

    return true;
}

bool testMGPUSubstractTrans1()
{
    int line = 4;
    int column = line + 1;
    int i = 1;
    int j = 2;
    float value = 2;
    float value2 = 1;
    float value3 = value - value2;
    float value4 = 4.5;
    float value5 = value - value4;

    MatrixGPU m1(line, column, value);
    MatrixGPU m2(column, line, value2);
    m2.set(i, j, value4);
    MatrixGPU m3(line, column, value3); 
    m3.set(j, i, value5);
    m1.transferGPU();
    m2.transferGPU();
    m1.subtractTrans(&m2);
    m1.transferCPU();
    return m1.isEqual(&m3);

}
bool testMGPUSubstractTrans2() {
    int line = 2;
    int column = line + 1;
    float value = 1.5;
    float value2 = 1;

    MatrixGPU m1(line, column, value);
    MatrixGPU m2(line, column, value2);
    MatrixGPU m3(column, line, value2);
    m3.transferGPU();
    try
    {
        m1.subtractTrans(&m2);
    }
    catch (std::invalid_argument&)
    {
        try
        {
            m1.subtractTrans(&m3);
        }
        catch (std::invalid_argument&)
        {
            return true;
        }
        return false;
    }

    return false;
}

bool testMGPUMultiply()
{
    int line = 2;
    int column = line + 1;
    float value = 1.5;
    float value2 = 1;
    float value3 = value * value2;
    MatrixGPU m1(line, column, value);
    MatrixGPU m3(line, column, value3);
    m1.transferGPU();
    m1.multiply(value2);
    m1.transferCPU();
    return m1.isEqual(&m3);
}

bool testMGPUMultiply2()
{
    std::chrono::high_resolution_clock::time_point t1;
    std::chrono::high_resolution_clock::time_point t2;
    int line = 4;
    int column = line + 1;
    float value = 1.5;
    float value2 = 2;
    float value3 = -3;

    MatrixGPU result(line, 1, 0, 1);

    MatrixGPU result2(line, 1, 0);
    MatrixGPU Mat(line, column, value);

    MatrixGPU vect(column, 1, value2);

    Mat.set(line - 2, column - 2, value3);
    vect.set(column - 3, 0, value3);

    try
    {
        result2.multiply(&Mat, &vect);

    }
    catch (const std::exception& e)
    {
        std::cout << e.what() << std::endl;
        return false;
    }

    Mat.transferGPU();
    vect.transferGPU();
   
    try
    {
        t1 = std::chrono::high_resolution_clock::now();
        result.multiply(&Mat, &vect);
        t2 = std::chrono::high_resolution_clock::now();
        std::cout << "temps de calcul de multiply su GPU " << std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count() << std::endl;
    }
    catch (const std::exception& e)
    {
        std::cout << e.what() << std::endl;
        return false;
    }
    
    
    result.transferCPU();
   
    bool testresult = result2.isEqual(&result);

    if (!testresult) {
        result.display();
        result2.display();
    }


    return testresult;
}

bool testMGPUMultiplyT1()
{
    int line = 2;
    int column = line + 1;
    float value = 1.5;
    float value2 = 1;
    float value3 = value * value2;

    MatrixGPU m1(line, column, value);
    MatrixGPU m2(line, column, value2);
    MatrixGPU m3(line, column, value3); 
    MatrixGPU temp(line, column);

    m1.transferGPU();
    m2.transferGPU();
    temp.transferGPU();

    temp.multiplyT(&m1, &m2);
    temp.transferCPU();

    return temp.isEqual(&m3);
}
bool testMGPUMultiplyT2()
{
    int line = 2;
    int column = line + 1;
    float value = 1.5;
    float value2 = 1;
    float value3 = value * value2;

    MatrixGPU m1(line, column, value);
    MatrixGPU m2(line, column, value2);
    MatrixGPU m3(line, column, value3); 
    m1.transferGPU();
    m2.transferGPU();
    
    m1.multiplyT(&m2);
    m1.transferCPU();

    return m1.isEqual(&m3);
}
bool testMGPUMultiplyT3()
{
    int line = 2;
    int column = line + 1;
    float value = 1.5;
    float value2 = 1;
    float value3 = value * value2;

    MatrixGPU m1(line, column, value);
    MatrixGPU m2(column, line, value2);
    MatrixGPU m3(line, column, value3);
    MatrixGPU temp(line, column);
    m1.transferGPU();
    m2.transferGPU();
    temp.transferGPU();

    try {
        temp.multiplyT(&m1, &m2);
    }
    catch (std::invalid_argument&) {
        try
        {
            temp.multiplyT(&m1, &m3);
        }
        catch (std::invalid_argument&)
        {
            return true;
        }
        return false;
    }
    return false;
}
bool testMGPUMultiplyT4()
{
    int line = 2;
    int column = line + 1;
    float value = 1.5;
    float value2 = 1;


    MatrixGPU m1(line, column, value);
    MatrixGPU m2(line, line, value2);
    MatrixGPU m3(line, column, value);
    m1.transferGPU();
    m2.transferGPU();

    try {
        m1.multiplyT(&m2);
    }
    catch (std::invalid_argument&) {

        try
        {
            m1.multiplyT(&m3);
        }
        catch (std::invalid_argument&)
        {
            return true;
        }
        return false;
    }
    return false;
}

bool testMGPUMultiplyVect()
{
    int nRow = 4;
    int nCol = 6;
    float value1 = 4;
    float value2 = 72.36;
    float value3 = -42.78;
    float value4 = 19.832;

    MatrixGPU y(nRow, 1, 0, 1);
    MatrixGPU yCPU(nRow, 1);

    MatrixGPU A(nRow, nCol, value1);
    MatrixGPU x(nCol, 1, value2);
    A.set(nRow - 1, nCol - 2, value3);
    x.set(nCol - 3, 0, value4);

    yCPU.multiply(&A, &x);
    A.transferGPU();
    x.transferGPU();

    y.multiply(&A, &x);
    y.transferCPU();


    return yCPU.isEqual(&y);
}

bool testMGPUMultiplyMat()
{
    int nRow = 4;
    int nCol = 6;
    int common = 5;
    float value1 = 4;
    float value2 = 72.36;
    float value3 = -42.78;
    float value4 = 19.832;

    MatrixGPU y(nRow, nCol, 0, 1);
    MatrixGPU yCPU(nRow, nCol);

    MatrixGPU A(nRow, common, value1);
    MatrixGPU x(common, nCol, value2);
    A.set(nRow - 1, common - 2, value3);
    x.set(common - 3, nCol - 4, value4);

    yCPU.multiplyMat(&A, &x);
    A.transferGPU();
    x.transferGPU();

    y.multiplyMat(&A, &x);
    y.transferCPU();


    return yCPU.isEqual(&y);
}

bool testMGPUMultiplyLinearOp()
{
    int nRow = 4;
    int nCol = 6;
    float value1 = 4;
    float value2 = 72.36;
    float value3 = -42.78;
    float value4 = 19.832;
    float value5 = 12.654;
    float value6 = 93.47;

    MatrixGPU y(nRow, 1, 0, 1);
    MatrixGPU yCPU(nRow, 1);

    MatrixGPU A(nRow, nCol, value1);
    MatrixGPU x(nCol, 1, value2);
    MatrixGPU b(nCol, 1, value5);
    A.set(nRow - 1, nCol - 2, value3);
    x.set(nCol - 3, 0, value4);
    b.set(nCol - 1, 0, value6);

    yCPU.linearOperation(&A, &x, &b);
    A.transferGPU();
    x.transferGPU();
    b.transferGPU();

    y.linearOperation(&A, &x, &b);
    y.transferCPU();


    return yCPU.isEqual(&y);
}



bool testMGPUDivide1()
{
    int line = 2;
    int column = line + 1;
    float value = 1.5;
    float value2 = 1.2;
    float value3 = value / value2;


    MatrixGPU m1(line, column, value);
    MatrixGPU m2(line, column, value2);
    MatrixGPU m3(line, column, value3); 
    m1.transferGPU();
    m2.transferGPU();
    m1.divideT(&m2);
    m1.transferCPU();
    return m1.isEqual(&m3);
}
bool testMGPUDivide2()
{
    int line = 2;
    int column = line + 1;
    float value = 1.5;
    float value2 = 1;


    MatrixGPU m1(line, column, value);
    MatrixGPU m2(column, line, value2);
    MatrixGPU m3(line, column, value);
    m3.transferGPU();
    try
    {
        m1.divideT(&m2);
    }
    catch (std::invalid_argument&)
    {
        try
        {
            m1.divideT(&m3);
        }
        catch (std::invalid_argument&)
        {
            return true;
        }
        return false;
    }
    return false;
}
bool testMGPUDivide3()
{
    int line = 2;
    int column = line + 1;
    float value = 1.5;
    float value2 = 1;


    MatrixGPU m1(line, column, value);
    MatrixGPU m2(line, column, value2);
    m2.set(line - 1, column - 1, 0);
    
    m1.transferGPU();
    m2.transferGPU();

    m1.divideT(&m2);
    m1.transferCPU();

    return (m1.get(line - 1, column - 1) == std::numeric_limits<float>::infinity());
    
}
bool testMGPUDivide4()
{
    int line = 2;
    int column = line + 1;
    float value = 1.5;
    float value2 = -1.2;
    float value3 = value / value2;
    MatrixGPU m1(line, column, value);
    MatrixGPU m3(line, column, value3);

    m1.transferGPU();

    m1.divide(value2);

    m1.transferCPU();

    return m1.isEqual(&m3);
}
bool testMGPUDivide5()
{
    int line = 2;
    int column = line + 1;
    float value = 1.5;
    float value2 = 0;


    MatrixCPU m1(line, column, value);


    try
    {
        m1.divide(value2);
    }
    catch (std::domain_error&)
    {
        return true;
    }
    return false;
}

bool testMGPUmoy1()
{
    int ligne = 3;
    int colonne = 4;
    float value = 1.5;
    MatrixGPU m1(ligne, colonne, 0);
    m1.set(0, 0, value);
    m1.set(0, 3, value);
    m1.set(1, 0, value);
    m1.set(1, 1, value);
    m1.set(1, 3, value);
    m1.set(2, 2, value);
    MatrixGPU nb1col(1, colonne, 0);
    nb1col.set(0, 0, 2);
    nb1col.set(0, 1, 1);
    nb1col.set(0, 2, 1);
    nb1col.set(0, 3, 2);
    MatrixGPU temp1(1, colonne);
    MatrixGPU m3(1, colonne, value);


    temp1.transferGPU();
    m1.transferGPU();
    nb1col.transferGPU();
    temp1.Moy(&m1, &nb1col, 1);
    temp1.transferCPU();

    return temp1.isEqual(&m3);
}
bool testMGPUmoy2()
{
    int ligne = 3;
    int colonne = 4;
    float value = 1.5;
    MatrixGPU m1(ligne, colonne, 0);
    m1.set(0, 0, value);
    m1.set(0, 3, value);
    m1.set(1, 0, value);
    m1.set(1, 1, value);
    m1.set(1, 3, value);
    m1.set(2, 2, value);
    MatrixGPU nb2li(ligne, 1, 2);
    MatrixGPU temp2(ligne, 1);
    MatrixGPU m5(ligne, 1, value);
    m5.set(1, 0, value * 3 / 2);
    m5.set(2, 0, value / 2);

    temp2.transferGPU();
    m1.transferGPU();
    nb2li.transferGPU();
    temp2.Moy(&m1, &nb2li, 0);
    temp2.transferCPU();
    

    return temp2.isEqual(&m5);
}
bool testMGPUmoy3()
{
    int ligne = 3;
    int colonne = 4;
    float value = 1.5;
    MatrixGPU m1(ligne, colonne, 0);
    m1.set(0, 0, value);
    m1.set(0, 3, value);
    m1.set(1, 0, value);
    m1.set(1, 1, value);
    m1.set(1, 3, value);
    m1.set(2, 2, value);
    MatrixGPU nb2li(ligne, 1, 2);
    MatrixGPU temp2(ligne, 1);
    try
    {
        temp2.Moy(&m1, &nb2li, 1); 
    }
    catch (std::invalid_argument&)
    {
        return true;
    }
    return false;
}
bool testMGPUmoy4()
{
    int ligne = 3;
    int colonne = 4;
    float value = 1.5;
    MatrixGPU m1(ligne, colonne, 0);
    m1.set(0, 0, value);
    m1.set(0, 3, value);
    m1.set(1, 0, value);
    m1.set(1, 1, value);
    m1.set(1, 3, value);
    m1.set(2, 2, value);
    MatrixGPU nb2li(ligne, 1, 2);
    MatrixGPU temp2(ligne, 1);
    MatrixGPU temp1(1, colonne);
    try
    {
        temp1.Moy(&m1, &nb2li, 0); 
    }
    catch (std::invalid_argument&)
    {
        try
        {
            temp2.Moy(&m1, &m1, 0); 
        }
        catch (std::invalid_argument&)
        {
            return true;
        }
        return false;
    }
    return false;
}

bool testMGPUProject1()
{
    int ligne = 3;
    int colonne = 4;
    float value = 1.5;
    float value2 = 0.5;
    float value3 = 3;
    MatrixGPU m1(ligne, colonne, value);
    m1.set(0, 2, value2 - 1);
    m1.set(1, 1, value3 + 1);
    MatrixGPU m2(ligne, colonne, value);
    m2.set(0, 2, value2);
    m2.set(1, 1, value3);
    MatrixGPU m22(m2);
    MatrixGPU Lb(ligne, colonne, value2);
    MatrixGPU Ub(ligne, colonne, value3);

    m1.transferGPU();
    m2.transferGPU();
    Lb.transferGPU();
    Ub.transferGPU();

    m1.project(&Lb, &Ub);
    m1.transferCPU();
    if (!m1.isEqual(&m22)) return false;
    m2.project(&Lb, &Ub);
    m2.transferCPU();
    if (!m2.isEqual(&m22)) return false;

    return true;
}
bool testMGPUProject2()
{
    int ligne = 3;
    int colonne = 4;
    float value = 1.5;
    float value2 = 0.5;
    float value3 = 3;
    MatrixGPU m1(ligne, colonne, value);
    MatrixGPU m3(colonne, ligne, value);
    MatrixGPU lb(ligne, colonne, value2);
    MatrixGPU ub(ligne, colonne, value3);

    lb.transferGPU();
    try
    {
        m1.project(&m3, &ub); 
    }
    catch (std::invalid_argument&)
    {
        try
        {
            m1.project(&lb, &ub); 
        }
        catch (std::invalid_argument&)
        {
            return true;
        }
        return false;
    }
    return false;

}

bool testMGPUProjectPos()
{
    int row = 2;
    int column = 2;
    float value = 1.5;
    float value2 = -0.5;

    MatrixGPU m1(row, column, value2);
    m1.set(0, 1, 0);
    m1.set(1, 1, value);
    MatrixGPU m2(row, column);
    m2.set(1, 1, value);
    m1.transferGPU();
    

    m1.projectPos();
    m1.transferCPU();

    return m1.isEqual(&m2);
}

bool testMGPUProjectNeg()
{
    int row = 2;
    int column = 2;
    float value = -1.5;
    float value2 = 0.5;

    MatrixGPU m1(row, column, value2);
    m1.set(0, 1, 0);
    m1.set(1, 1, value);
    MatrixGPU m2(row, column);
    m2.set(1, 1, value);


    m1.transferGPU();


    m1.projectNeg();
    m1.transferCPU();


    return m1.isEqual(&m2);
}



bool testMGPUSum1()
{
    int ligne = 100;
    int colonne = 200;
    float value = 1.5;
    float value3 = 5;
    float value2 = (ligne * colonne -1) * value+value3;
    MatrixGPU m1(ligne, colonne, value);
    MatrixGPU m11(ligne, colonne, value);
    m1.set(ligne - 1, colonne - 1, value3);
    m11.set(ligne - 1, colonne - 1, value3);
    m1.transferGPU();
    float value4 = m11.sum();
    float value5 = m1.sum();

    std::cout << "testMGPUSum1 : " << value2 << " " << value4 << " " << value5 << std::endl;
    return ((value2 == value4) && (value2==value5));
}
bool testMGPUSum2()
{
    int ligne = 3;
    int colonne = 4;
    float value = 1.5;
    float value2 = ligne * colonne * value;
    MatrixGPU m1(ligne, colonne, value);
    MatrixGPU temp2(ligne, 1);
    MatrixGPU m4(ligne, 1, value * colonne);

    m1.transferGPU();
    temp2.transferGPU();

    temp2.sum(&m1);
    temp2.transferCPU();

    return temp2.isEqual(&m4);
}
bool testMGPUSum3()
{
    int ligne = 3;
    int colonne = 4;
    float value = 1.5;
    float value2 = ligne * colonne * value;
    MatrixCPU m1(ligne, colonne, value);
    MatrixCPU m2(colonne, colonne, value);
    MatrixCPU temp2(ligne, 1);

    try
    {
        temp2.sum(&m2, 0); 
    }
    catch (std::invalid_argument&)
    {
        try
        {
            m1.sum(&m2, 0); 
        }
        catch (std::invalid_argument&)
        {
            return true;
        }
        return false;
    }
    return false;
}
bool testMGPUSum4()
{
    int ligne = 3;
    int colonne = 4;
    float value = 1.5;
    float value2 = ligne * colonne * value;
    MatrixGPU m1(ligne, colonne, value);
    MatrixGPU temp2(ligne, 1);

    temp2.transferGPU();
    try
    {
        temp2.sum(&m1); 
    }
    catch (std::invalid_argument&)
    {
        return true;
    }
    return false;
}

bool testMGPUSumPartial()
{
    int ligne = 200;
    int colonne = 100;
    float value = 1.5;
    float value3 = 5;
    float value2 = (ligne * colonne) * value /2 ;
    float value22 = ((ligne * colonne) / 2 - 1)* value + value3;
    MatrixGPU m1(ligne, colonne, value);
    MatrixGPU m11(ligne, colonne, value);
    m1.set(ligne - 1, colonne - 1, value3);
    m11.set(ligne - 1, colonne - 1, value3);
    m1.transferGPU();
    float value4 = m11.sum(0, ligne * colonne / 2);
    float value5 = m11.sum(ligne * colonne / 2, ligne * colonne);

    float value42 = m1.sum(0, ligne * colonne / 2);
    float value52 = m1.sum(ligne * colonne / 2, ligne * colonne);

    std::cout << "testMGPUSumPartial : " << value2  << " " << value4 << " " << value42 << std::endl;
    std::cout << "testMGPUSumPartial : " << value22 << " " << value5 << " " << value52 << std::endl;
    return ((value2 == value4) && (value2 == value42) && (value22 == value5) && (value22 == value52));
}

bool testMGPUSwap() {
    int line = 100;
    int column = line + 1;
    float value = 1.5;
    float value2 = 1;
    float value3 = value + value2;

    MatrixGPU m1(line, column, value);
    MatrixGPU m11(line, column, value);
    MatrixGPU m2(line, column, value2);
    MatrixGPU m22(line, column, value2);

    m1.transferGPU();
    m2.transferGPU();
    m1.swap(&m2);

    m1.transferCPU();
    m2.transferCPU();

    return ((m1.isEqual(&m22)) && (m2.isEqual(&m11)));

}

bool testMGPUDistance()
{
    int colonne = 8;
    int ligne = 8;
    float value1 = 2;
    MatrixGPU m(ligne, colonne, value1);
    float value2 = sqrtf(ligne * colonne * value1 * value1);
    float value4 = m.distance2();

    if (value4 != value2) return false;

    m.transferGPU();
    float value3 = m.distance2();


    return (value2==value3);
}

bool testMGPUDistance2()
{
    int colonne = 2345;
    int ligne = 1234;
    float value1 = 2;
    float value5 = -5;
    MatrixGPU m(ligne, colonne, value1);
    MatrixGPU m2(ligne, colonne, value5);
    float value2 = sqrtf(ligne * colonne * (value1-value5) * (value1 - value5));
    float value4 = m.distance2(&m2);


    std::cout << "distance " << value2 << " " << value4 << std::endl;


    if (value4 != value2) return false;

    m.transferGPU();
    m2.transferGPU();
    float value3 = m.distance2(&m2);

    std::cout << "distance " << value2 << " " << value3 << std::endl;
    return (value2 == value3);
}

bool testMGPUMax()
{
    int colonne = 1;
    int ligne = 1;
    float value1 = 2;
    float value2 = 4;
    MatrixGPU m(ligne, colonne, value1);
    m.set(ligne - 1, colonne - 1, value2);

    float value3 = m.max2();
    


    if (value3 != value2) return false;

    m.transferGPU();

    float value4 = m.max2();

    std::cout << "testMGPUMax : " << value2 << " " << value3 << " " << value4 << std::endl;
    return (value2 == value4);
}

bool testMGPUMax2()
{
    int colonne = 1000;
    int ligne = 1000;
    float value1 = 2;
    float value2 = 4;
    float value3 = -4;
    MatrixGPU m(ligne, colonne, value1);
    m.set(ligne - 2, colonne - 2, value2);
    MatrixGPU m2(ligne, colonne, value3);
    

    float value4 = m.max2(&m2);


    if (value4 != (value2-value3)) return false;

    m.transferGPU();
    m2.transferGPU();
    value4 = m.max2(&m2);

    
    return ((value2 - value3) == value4);
}


bool testMGPUMax3()
{
    int colonne = 1000;
    int ligne = 1000;
    float value1 = 2;
    float value2 = -4;
    MatrixGPU m(ligne, colonne, value1);
    m.set(ligne - 2, colonne - 2, value2);


    int pos1 = (ligne - 2) * colonne + colonne - 2;
    int pos2 = 0;
    float value3 = m.max2(&pos2);
    

    if ((abs(value3) != abs(value2))||(pos1 != pos2)) return false;

    m.transferGPU();

    float value4 = m.max2();

   
    return ((abs(value2) == abs(value4)) && (pos1 == pos2));
}

bool testMGPUDivideGJ1()
{
    int n = 3;
    MatrixGPU ident(n, n, 0, 1);
    ident.setEyes(1);

    MatrixGPU invert(n, n, 0, 1);
    invert.invertGaussJordan(&ident);
    
    ident.transferCPU();
    invert.transferCPU();

    if (!ident.isEqual(&invert)) return false;
   

    MatrixGPU m1(n, n);
    m1.set(0, 0, 2);
    m1.set(0, 1, -1);
    m1.set(1, 1, -1);
    m1.set(1, 2, 2);
    m1.set(2, 0, -1);
    m1.set(2, 1, 2);
    m1.set(2, 2, 1);
    m1.display();
    m1.transferGPU();

    MatrixGPU m2(n, n, 0,1);
    MatrixGPU m22(n, n);
    m22.set(0, 0, 5.0 / 8);
    m22.set(0, 1, -1.0 / 8);
    m22.set(0, 2, 1.0 / 4);
    m22.set(1, 0, 1.0 / 4);
    m22.set(1, 1, -1.0 / 4);
    m22.set(1, 2, 1.0 / 2);
    m22.set(2, 0, 1.0 / 8);
    m22.set(2, 1, 3.0 / 8);
    m22.set(2, 2, 1.0 / 4);
   
    m2.invertGaussJordan(&m1);

    m2.transferCPU();

    m22.display();
    m2.display();
    //
    MatrixCPU m4(5, 5);
    
    m4.set(0, 0, 1.5);
    m4.set(0, 4, 1);
    m4.set(1, 1, 1.5);
    m4.set(1, 2, 1);
    m4.set(1, 3, -1);
    m4.set(1, 4, 1);
    m4.set(2, 1, -0.00000002829688838801303063519299030303955078125);
    m4.set(3, 1, 0.00000002829688838801303063519299030303955078125);
    m4.set(2, 2, -0.800027906894683837890625);
    m4.set(3, 3, -0.799972116947174072265625);
    m4.set(4, 0, 1);
    m4.set(4, 1, 1);
    m4.display();

    MatrixGPU m4GPU(m4,1);

    MatrixCPU m3(5,5);
    MatrixGPU m3GPU(5, 5, 0, 1);
    MatrixCPU m3CPU(5, 5);
    
    m3.invertGaussJordan(&m4);
    m3GPU.invertGaussJordan(&m4GPU);
    
    m3GPU.toMatCPU(m3CPU);

    m3CPU.display();
    m3.display();

    return m2.isEqual(&m22) && m3CPU.isEqual(&m3);
}

bool testMGPUDivideGJ2()
{
    int n = 3;
    MatrixGPU temp1(n, n + 1);
    MatrixGPU temp2(n, n);
    temp2.set(0, 0, 1);
    temp2.set(2, 2, 1);

    //temp2.transferGPU();
    //temp1.transferGPU();

    try
    {
        temp1.invertGaussJordan(&temp1); // not square matrix
    }
    catch (std::invalid_argument&)
    {
        try
        {
            temp1.invertGaussJordan(&temp2); // not same size
        }
        catch (std::invalid_argument&)
        {
            try
            {
                temp2.invertGaussJordan(&temp2); // not invertible
            }
            catch (std::invalid_argument&)
            {
                return true;
            }
            return false;
        }
        return false;
    }
    return false;
}

bool testMGPUSolveSys()
{
    Eigen::Matrix3f M;
    Eigen::Vector3f b;
    M << 1, 2, 3, 4, 5, 6, 7, 8, 10;
    b << 3, 3, 4;
    std::cout << "Here is the matrix M:\n" << M << std::endl;
    std::cout << "Here is the vector b:\n" << b << std::endl;
    Eigen::Vector3f x = M.colPivHouseholderQr().solve(b);
    std::cout << "The solution is:\n" << x << std::endl;

    MatrixGPU Mm(3, 3);
    MatrixGPU bm(3, 1);
    MatrixGPU xm(3, 1, 0, 1);
    MatrixGPU Am(3, 3, 0, 1);
    MatrixGPU P(4, 1, 0, 1);

    MatrixGPU Mm2(3, 3);
    MatrixGPU bm2(3, 1);
    MatrixGPU xm2(3, 1);
    MatrixGPU Am2(3, 3);
    MatrixGPU P2(4, 1);

    for (int i = 0; i < 3; i++) {
        bm.set(i, 0, b(i));
        bm2.set(i, 0, b(i));
        for (int j = 0; j < 3; j++) {
            Mm.set(i, j, M(i, j));
            Mm2.set(i, j, M(i, j));
        }
    }
    /*std::cout << "M sur CPU" << std::endl;
    Mm2.display();
    std::cout << "M sur GPU" << std::endl;
    Mm.display();
    std::cout << "b sur CPU" << std::endl;
    bm2.display();
    std::cout << "b sur GPU" << std::endl;
    bm.display();*/

    Mm.transferGPU();
    bm.transferGPU();

    Mm.LUPFactorization(&Am, &P);
    Mm2.LUPFactorization(&Am2, &P2);
    std::cout << "A sur CPU" << std::endl;
    Am2.display();
    std::cout << "A sur GPU" << std::endl;
    Am.display(true);
    std::cout << "P sur CPU" << std::endl;
    P2.display();
    std::cout << "P sur GPU" << std::endl;
    P.display(true);/**/

    
    xm.solveSys(&Am, &P, &bm);
    xm2.solveSys(&Am2, &P2, &bm2);
   
    xm.transferCPU();
    

    /*std::cout << "solution sur CPU" << std::endl;
    xm2.display();
    std::cout << "solution sur GPU" << std::endl;
    xm.display();*/

    for (int i = 0; i < 3; i++) {
        if (abs(xm.get(i, 0) - x(i)) > 0.00001) {
            std::cout << xm.get(i, 0) << " " << x(i) << " " << std::endl;
            return false;
        }
    }
    for (int i = 0; i < 3; i++) {
        if (abs(xm2.get(i, 0) - x(i)) > 0.00001) {
            std::cout << xm2.get(i, 0) << " " << x(i) << " " << std::endl;
            return false;
        }
    }


    return true;
}
