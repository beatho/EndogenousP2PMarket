#include "hip/hip_runtime.h"


#include "../head/Utilities.cuh"
#define PI 3.14159265359
#define ITERNEWTON 50
#define EPSNEWTON 0.00000001
#define F3(a, b, c, x) (x*x*x + a *x*x + b*x + c)
#define F4(a, b, c, d, x) (x*x*x*x + a *x*x*x + b*x*x + c*x + d)

#define FPRIM3(a, b, x) (3*x*x + 2*a*x + b)
#define FPRIM4(a, b, c, x) (4*x*x*x + a*3*x*x + b*2*x + c)

#define FSECON3(a, x) (6*x + 2*a)
#define FSECON4(a, b, x) (12*x*x + a*6*x + b*2)

#define MAX(X, Y) X * (X >= Y) + Y * (Y > X)

template <typename T>
void check(T err, const char* const func, const char* const file,
	const int line)
{
	if (err != hipSuccess)
	{
		std::cerr << "CUDA Runtime Error at: " << file << ":" << line
			<< std::endl;
		std::cerr << hipGetErrorString(err) << " " << func << std::endl;
		// We don't exit when we encounter CUDA errors in this example.
		// std::exit(EXIT_FAILURE);
	}
}


void checkLast(const char* const file, const int line)
{
	hipDeviceSynchronize();
	hipError_t err{ hipGetLastError() };
	if (err != hipSuccess)
	{
		std::cerr << "CUDA Runtime Error at: " << file << ":" << line
			<< std::endl;
		std::cerr << hipGetErrorString(err) << std::endl;
		// We   exit when we encounter CUDA errors in this example.
		std::exit(EXIT_FAILURE);
	}
}






int resolveRealPolynome3without2term(double* root, double* coef) {
	/*
	* return : the number of real root for the polynome x^3 + px + q = 0
	* root : is a array of size 3
	* coeff : is a array of size 2 (p and q of x^3 + px + q = 0)
	* if complexe root, root[1] is the real part and root[2] the imaginary part 
	* must add -b/3a if bx^2 is not null at the begining and coefPolynome3From4to2coef is used
	*/
	double p = coef[0];
	double q = coef[1];
	double Delta = -4 * p*p*p - 27 *q*q;
	if (Delta == 0) {
		root[0] = -3 * q / (2 * p);
		root[1] = -2 * root[0];
		root[2] = root[0];
		return 2;
	}
	else if (Delta < 0) {
		double z0 = cbrt((-q + sqrt(-Delta / 27.0)) / 2.0) + cbrt((-q - sqrt(-Delta / 27.0)) / 2.0);
		root[0] = z0; // b2 = z0

		double c2 = p + z0 * z0;
		double delta2 = z0 * z0 - 4 * c2; // n�gatif normalement
		root[1] = -z0 / 2; // partie r�elle de la racine double
		root[2] = sqrt(-delta2) / 2;
		return 1;
	}
	else {
		double r = (3 * q * sqrt(3)) / (2 * p * sqrt(-p)) ;
		for (int k = 0; k < 3; k++) {
			root[k] = 2.0 * sqrt(-p / 3.0) * cos((acos(r) + 2.0 * k * PI) / 3.0	);
		}

		return 3;
	}
}

int resolveRealPolynome3Newton(double* root, double* coef, double init)
{
	/*
	* return : the number of real root for the polynome x^3 + s x^2 + px + q = 0
	* root : is a array of size 3
	* coeff : is a array of size 3 (s and p and q of x^3 + s x^2 + px + q = 0 )
	*/
	
	double x_i = 0;
	int nRoot = 0;
	int i = 0;
	double x_pre = init;
	if (coef[2] != 0) {
		// solve Newton
				
		double eps = 2 * EPSNEWTON;

		while (i<ITERNEWTON && eps>EPSNEWTON)
		{
			//std::cout << i << " : " << x_i << std::endl;
			x_i = x_pre - (F3(coef[0], coef[1], coef[2], x_pre)) / (FPRIM3(coef[0], coef[1], x_pre));
			eps = (x_i - x_pre) * (x_i - x_pre);
			x_pre = x_i;
			i++;
		}	
		//std::cout << i << " " << eps << " " << x_i << std::endl;
		
	}
	if (i == ITERNEWTON) {
		i = 0; // on r�essaie avec une autre init
		double b = coef[0];
		double c = coef[1];
		double d = coef[2];

		double p = (3 * c - b * b) / (3);
		double q = (2 * b * b * b - 9 * b * c + 27 * d) / (27);

		if (q == 0) {
			x_i = 0 - b / 3;
		}
		else if (F3(0, p, q, init) > 0) {
			x_pre = findAntpoly3Neg(p, q);
			if (F3(0, p, q, x_pre) >= 0) {
				std::cout << "polynome " << p << " " << q << std::endl;
				std::cout << "probl�me on the solution neg " << x_pre << " " << F3(0, p, q, x_pre) << std::endl;
			}
		}
		else {
			x_pre = findAntpoly3Pos(p, q);
			if (F3(0, p, q, x_pre) <= 0) {
				std::cout << "polynome " << p << " " << q << std::endl;
				std::cout << "probl�me on the solution pos " << x_pre << " " << F3(0, p, q, x_pre) << std::endl;
			}
		}
		double eps = 2 * EPSNEWTON;

		while (i<ITERNEWTON && eps>EPSNEWTON)
		{
			//std::cout << i << " : " << x_i << std::endl;
			x_i = x_pre - (F3(0, p, q, x_pre)) / (FPRIM3(0, p, x_pre));
			eps = (x_i - x_pre) * (x_i - x_pre);
			x_pre = x_i;
			i++;
		}
		x_i = x_i - b / 3;

		if (i == ITERNEWTON) { // racine non trouv� ...
			std::cout << "***************** Cela n'a pas march� **********" << std::endl;
			double coefTemp[2];

			coefTemp[0] = p;
			coefTemp[1] = q;
			nRoot = resolveRealPolynome3without2term(root, coefTemp);
			std::cout << "analytics method used, nRoot = " << nRoot << std::endl;
			for (int k = 0; k < nRoot; k++) {
				root[k] = root[k] - b / 3;
			}

			return nRoot;
		}
	}
	
	root[nRoot] = x_i;
	nRoot++;
	
	// second degr�
	// x^2 + b x + c = 0 tel que  x^3 + s x^2 + px + q = (x-x_i)(x^2 + b x + c)
	double B = coef[0] + x_i;
	double C = coef[1] + x_i * B;


	double delta = B * B - 4 * C;
	
	if (delta == 0) {
		double z = -B / 2;
		root[nRoot] = z;
		nRoot++;
		//root[1] = z;
		//nRoot++;
		//std::cout << " z " << z << std::endl;
		return nRoot;
	}
	else if (delta > 0) {
		double z1 = (-B + sqrt(delta)) / 2;
		double z2 = (-B - sqrt(delta)) / 2;
		root[nRoot] = z1;
		nRoot++;
		root[nRoot] = z2;
		nRoot++;
		return nRoot;
	}
	else { // delta < 0
		//std::cout << "pas d'autres racines r�elle !!!! " << std::endl;
	}
return nRoot;
	
	
	
}


int resolveRealPolynome3Laguerre(double* root, double* coef, double init)
{
	/*
	* return : the number of real root for the polynome x^3 + s x^2 + px + q = 0
	* root : is a array of size 3
	* coeff : is a array of size 3 (s and p and q of x^3 + s x^2 + px + q = 0 )
	*/

	double x_i = 0;
	int nRoot = 0;
	int n = 3; // degr� du polynome
	int i = 0;
	if (coef[2] != 0) {
		// solve Laguerre
		double x_pre = init;

		
		double eps = 2 * EPSNEWTON;

		while (i<ITERNEWTON && eps>EPSNEWTON)
		{
			std::cout << i << " : " << x_i << std::endl;
			double p = F3(coef[0], coef[1], coef[2], x_pre);
			double p2 = FPRIM3(coef[0], coef[1], x_pre);
			double p3 = FSECON3(coef[0], x_pre);
			double S1 = p / p2;
			double S2 = p3 / p - S1 * S1;

			if (S1 > 0) {
				x_i = x_pre - n / (S1 + sqrt((1 - n) * (n * S2 + S1 * S1)));
			}
			else {
				x_i = x_pre - n / (S1 - sqrt((1 - n) * (n * S2 + S1 * S1)));
			}

			eps = (x_i - x_pre) * (x_i - x_pre);
			x_pre = x_i;
			i++;
		}
		std::cout << i << " " << eps << " " << x_i << std::endl;
	}
	if (i == ITERNEWTON) { // racine non trouv� ...
		double coefTemp[2];
		double b = coef[0];
		double c = coef[1];
		double d = coef[2];

		double p = (3 * c - b * b) / (3);
		double q = (2 * b * b * b - 9 * b * c + 27 * d) / (27);
		coefTemp[0] = p;
		coefTemp[1] = q;
		nRoot = resolveRealPolynome3without2term(root, coefTemp);
		std::cout << "analytics method used, nRoot = " << nRoot << std::endl;
		for (int k = 0; k < nRoot; k++) {
			root[k] = root[k] - b / 3;
		}

		return nRoot;
	} 

	root[nRoot] = x_i;
	nRoot++;

	// second degr�
	// x^2 + b x + c = 0 tel que  x^3 + s x^2 + px + q = (x-x_i)(x^2 + b x + c)
	double B = coef[0] + x_i;
	double C = coef[1] + x_i * B;


	double delta = B * B - 4 * C;

	if (delta == 0) {
		double z = -B / 2;
		root[nRoot] = z;
		nRoot++;
		//root[1] = z;
		//nRoot++;
		//std::cout << " z " << z << std::endl;
		return nRoot;
	}
	else if (delta > 0) {
		double z1 = (-B + sqrt(delta)) / 2;
		double z2 = (-B - sqrt(delta)) / 2;
		root[nRoot] = z1;
		nRoot++;
		root[nRoot] = z2;
		nRoot++;
		return nRoot;
	}
	else { // delta < 0
		//std::cout << "pas d'autres racines r�elle !!!! " << std::endl;
	}


	return nRoot;
}


int resolveRealPolynome3Halley(double* root, double* coef, double init)
{
	/*
	* return : the number of real root for the polynome x^3 + s x^2 + px + q = 0
	* root : is a array of size 3
	* coeff : is a array of size 3 (s and p and q of x^3 + s x^2 + px + q = 0 )
	*/

	double x_i = 0;
	int nRoot = 0;
	int n = 3; // degr� du polynome
	int i = 0;
	double x_pre = init;
	
	if (coef[2] != 0) {
		// solve Laguerre
		
		double eps = 2 * EPSNEWTON;

		while (i<ITERNEWTON && eps>EPSNEWTON)
		{
			//std::cout << i << " : " << x_i << std::endl;
			double p = F3(coef[0], coef[1], coef[2], x_pre);
			double p2 = FPRIM3(coef[0], coef[1], x_pre);
			double p3 = FSECON3(coef[0], x_pre);
			
			x_i = x_pre - (2 * p * p2) / (2 * p2 * p2 - p * p3);

			eps = (x_i - x_pre) * (x_i - x_pre);
			x_pre = x_i;
			i++;
		}
	}
	if (i == ITERNEWTON) {
		i = 0; // on r�essaie avec une autre init
		double b = coef[0];
		double c = coef[1];
		double d = coef[2];

		double p = (3 * c - b * b) / (3);
		double q = (2 * b * b * b - 9 * b * c + 27 * d) / (27);

		if (q == 0) {
			x_i = 0 - b / 3;
		}
		else if (F3(0, p, q, init) > 0) {
			x_pre = findAntpoly3Neg(p, q);
			if (F3(0, p, q, x_pre) >= 0) {
				std::cout << "polynome " << p << " " << q << std::endl;
				std::cout << "probl�me on the solution neg " << x_pre << " " << F3(0, p, q, x_pre) << std::endl;
			}
		}
		else {
			x_pre = findAntpoly3Pos(p, q);
			if (F3(0, p, q, x_pre) <= 0) {
				std::cout << "polynome " << p << " " << q << std::endl;
				std::cout << "probl�me on the solution pos " << x_pre << " " << F3(0, p, q, x_pre) << std::endl;
			}
		}
		double eps = 2 * EPSNEWTON;

		while (i<ITERNEWTON && eps>EPSNEWTON)
		{
			//std::cout << i << " : " << x_i << std::endl;
			double p1 = F3(0, p, q, x_pre);
			double p2 = FPRIM3(0, p, x_pre);
			double p3 = FSECON3(0, x_pre);

			x_i = x_pre - (2 * p1 * p2) / (2 * p2 * p2 - p1 * p3);

			eps = (x_i - x_pre) * (x_i - x_pre);
			x_pre = x_i;
			i++;
		}
		x_i = x_i - b / 3;

		if (i == ITERNEWTON) { // racine non trouv� ...
			std::cout << "***************** Cela n'a pas march� **********" << std::endl;
			double coefTemp[2];
			
			coefTemp[0] = p;
			coefTemp[1] = q;
			nRoot = resolveRealPolynome3without2term(root, coefTemp);
			std::cout << "analytics method used, nRoot = " << nRoot << std::endl;
			for (int k = 0; k < nRoot; k++) {
				root[k] = root[k] - b / 3;
			}

			return nRoot;
		}
	}
	
	
	
	

	root[nRoot] = x_i;
	nRoot++;

	// second degr�
	// x^2 + b x + c = 0 tel que  x^3 + s x^2 + px + q = (x-x_i)(x^2 + b x + c)
	double B = coef[0] + x_i;
	double C = coef[1] + x_i * B;


	double delta = B * B - 4 * C;

	if (delta == 0) {
		double z = -B / 2;
		root[nRoot] = z;
		nRoot++;
		//root[1] = z;
		//nRoot++;
		//std::cout << " z " << z << std::endl;
		return nRoot;
	}
	else if (delta > 0) {
		double z1 = (-B + sqrt(delta)) / 2;
		double z2 = (-B - sqrt(delta)) / 2;
		root[nRoot] = z1;
		nRoot++;
		root[nRoot] = z2;
		nRoot++;
		return nRoot;
	}
	else { // delta < 0
		//std::cout << "pas d'autres racines r�elle !!!! " << std::endl;
	}


	return nRoot;
}


int resvolveRealPolynome4without2term(double* root, double* coef)
{
	/*
	* return : the number of real root for the polynome x^4 + bx^3 + dx + e = 0
	* root : is a array of size 4
	* coeff : is a array of size 3 (b,d,e)
	*/
	double b = coef[0];
	double d = coef[1];
	double e = coef[2];
	int nRoot = 0;

	if (b * b * b + 8 * d == 0) {
		//if (abs(b * b * b + 8 * d) < 0.00000001) {
		// passage de p^4 + b p^3 + d p + e -> a p^4 + b p^2 + c = 0
		double B = -3 * b * b / 8;
		double C = -3* b*b*b*b/256 - b*d/4 + e;

		double delta = B * B - 4 * C;
		//std::cout << "Delta " << delta;
		if (delta  == 0) {
			double z = - B / 2;
			nRoot = 2;
			//std::cout << " z " << z << std::endl;
			root[0] = sqrt(z);
			root[1] = -sqrt(z);
			return nRoot;
		}
		else if (delta > 0) {
			double z1 = (-B + sqrt(delta)) / 2;
			double z2 = (-B - sqrt(delta)) / 2;
			//std::cout << " z1 " << z1 << " z2 " << z2 << std::endl;
			if (z1 >= 0) {
				root[0] = sqrt(z1);
				root[1] = -sqrt(z1);
				nRoot = 2;
			} if (z2 >= 0) {
				root[nRoot] = sqrt(z2);
				root[nRoot + 1] = -sqrt(z2);
				nRoot += 2;
			}
			return nRoot;
		}
		else { // delta < 0
			//std::cout << "pas de racines r�elle !!!! rip, on tente le pas bicarr�" << std::endl;
		}
	}

	// for the lambda polynome
	double coef2[2];
	double rootlambda[3];
	coef2[0] = (2 * b * d - 8 * e) / 8;
	coef2[1] = -(b * b * e + d * d) / 8;
	int nRootlambda = resolveRealPolynome3without2term(rootlambda, coef2);


	
	
	for (int i = 0; i < nRootlambda; i++) {
		double lambda0 = rootlambda[i];
		//std::cout << "poly3 " << coef2[0] * lambda0 + coef2[1] + lambda0 * lambda0 * lambda0 << std::endl;
		double mu1 = 2 * lambda0 + (b * b) / 4;
		if (mu1 > 0) {
			double mu0 = sqrt(mu1);
			double DeltaP = -2 * lambda0 + 2 * (d - b * lambda0) / mu0 + b * mu0 + b * b / 2;
			double DeltaM = -2 * lambda0 - 2 * (d - b * lambda0) / mu0 - b * mu0 + b * b / 2;
			if (DeltaP >= 0) {
				root[nRoot] = (-mu0 + sqrt(DeltaP)) / 2 - b / 4;
				root[nRoot + 1] = (-mu0 - sqrt(DeltaP)) / 2 - b / 4;
				nRoot = nRoot + 2;
				//std::cout << "  Dp   ";
			}
			if (DeltaM >= 0) {
				root[nRoot] = (mu0 + sqrt(DeltaM)) / 2 - b / 4;
				root[nRoot + 1] = (mu0 - sqrt(DeltaM)) / 2 - b / 4;
				nRoot = nRoot + 2;
				//std::cout << "  DM   ";
			}
			if (nRoot > 0) {
				//std::cout << "poly4 " << coef[0] << " " <<  coef[1] << " " << coef[2] << std::endl;
				return nRoot;
			}
		}
		
	}
	double lambda0 = rootlambda[0];
	double mu0 = sqrt(2 * lambda0 + (b * b) / 4);
	double DeltaP = -2 * lambda0 + 2 * (d - b * lambda0) / mu0 + b * mu0 + b * b / 2;
	double DeltaM = -2 * lambda0 - 2 * (d - b * lambda0) / mu0 - b * mu0 + b * b / 2;
	//std::cout << "no real root found " << abs(b * b * b + 8 * d) << " " << lambda0 << " " << mu0 << " " << DeltaP << " " << DeltaM << std::endl;
	
	return nRoot;
}

void coefPolynome3From4to2coef(double* coef4, double* coef2)
{

	double a = coef4[0];

	if (a == 0) {
		throw std::invalid_argument("must be a thrid degree polynome, a!=0 ");
	}

	double b = coef4[1];
	double c = coef4[2];
	double d = coef4[3];

	coef2[0] = (3 * a * c - b * b) / (3 * a * a);
	coef2[1] = (2 * b * b * b - 9 * a * b * c + 27 * a * a * d) / (27 * a * a * a);

}


int resolveRealPolynome4Newton(double* root, double* coef, double init)
{
	/*
	* return : the number of real root for the polynome x^4 + bx^3 + c x^2  + dx + e = 0
	* root : is a array of size 4
	* coeff : is a array of size 4 (b, c, d,e)
	*/
	double coef3[3];
	double x_i = 0;
	
	if (coef[3] != 0) {
		// solve Newton
		double x_pre = init;

		int i = 0;
		double eps = 2 * EPSNEWTON;

		while (i<ITERNEWTON && eps>EPSNEWTON)
		{
			//std::cout << i << " : " << x_i << std::endl;
			x_i = x_pre - (F4(coef[0], coef[1], coef[2], coef[3], x_pre)) / (FPRIM4(coef[0], coef[1], coef[2], x_pre));
			eps = (x_i - x_pre) * (x_i - x_pre);
			x_pre = x_i;
			i++;
		}
	}
	
	// troisi�me degr�
	// x^2 + b x + c = 0 tel que  x^3 + s x^2 + px + q = (x-x_i)(x^2 + b x + c)
	coef3[0] = coef[0] + x_i;
	coef3[1] = coef[1] + x_i * coef3[0];
	coef3[2] = coef[2] + x_i * coef3[1];

	//std::cout << "Nouveau Polynome " << coef3[0] << " " << coef3[1] << " " << coef3[2] << std::endl;

	int nRoot = resolveRealPolynome3Newton(root, coef3, 0);
	root[nRoot] = x_i;
	nRoot++;

	return nRoot;
}


int resolveRealPolynome4Halley(double* root, double* coef, double init)
{
	/*
	* return : the number of real root for the polynome x^4 + bx^3 + c x^2  + dx + e = 0
	* root : is a array of size 4
	* coeff : is a array of size 4 (b, c, d,e)
	*/

	double x_i = 0;

	int n = 3; // degr� du polynome
	if (coef[2] != 0) {
		 
		double x_pre = init;

		int i = 0;
		double eps = 2 * EPSNEWTON;

		while (i<ITERNEWTON && eps>EPSNEWTON)
		{
			double p = F4(coef[0], coef[1], coef[2], coef[3], x_pre);
			double p2 = FPRIM4(coef[0], coef[1], coef[2], x_pre);
			double p3 = FSECON4(coef[0], coef[1], x_pre);

			x_i = x_pre - (2 * p * p2) / (2 * p2 * p2 - p * p3);

			eps = (x_i - x_pre) * (x_i - x_pre);
			x_pre = x_i;
			i++;
		}
	}


	// troisi�me degr�
	double coef3[3];


	double b = coef[0] + x_i;
	double c = coef[1] + x_i * b;
	double d = coef[2] + x_i * c;

	/*double p = (3  * c - b * b) / (3);
	//double q = (2 * b * b * b - 9  * b * c + 27  * d) / (27);
	//coef3[0] = 0;
	//coef3[1] = p;
	//coef3[2] = q;
	std::cout << "Nouveau Polynome " << coef3[0] << " " << coef3[1] << " " << coef3[2] << std::endl;
	std::cout << "Nouveau Polynome ou " << b << " " << c << " " << d << std::endl;*/
	coef3[0] = b;
	coef3[1] = c;
	coef3[2] = d;

	int nRoot = resolveRealPolynome3Halley(root, coef3, 0);
	/*for (int i = 0; i < nRoot; i++) {
		root[i] = root[i] - b / 3;
	}*/
	root[nRoot] = x_i;
	nRoot++;
	

	return nRoot;
}

int resvolveRealPolynome4without2termLagrange(double* root, double* coef) {
	/*
	* return : the number of real root for the polynome x^4 + bx^3 + dx + e = 0
	* root : is a array of size 4
	* coeff : is a array of size 3 (b,d,e)
	*/
	double b = coef[0];
	double d = coef[1];
	double e = coef[2];
	int nRoot = 0;

	// il faut passer de b d e � p q t, c'est le coef devant z^3 qui doit etre nul pas celui devant z^2 !!!

	double p = -3.0 * b * b / (8.0);
	double q = d + (b * b * b) / 8.0;
	double t = e - b * d / 4.0 - 3 * b * b * b * b / 256;

	//t = e/a - b*d/(4*a^2) + c*b^2/(16*a^3) - 3*b^4/(256 * a^4);


	int signe = 1 - 2 * (q > 0);

	/*double b2 = 2 * p;
	double c2 = (p * p - 4 * t);
	double d2 = -q * q;

	std::cout << b2 << " " << c2 << " " << d2 << std::endl;*/

	double coef2[2];
	double rootlambda[3];
	//coef2[0] = -b2*b2 / 3.0 + c2;
	//coef2[1] = (b2 / 27.0) * (2 * b2 *b2 - 9 * c2) + d2;
	coef2[0] = -4.0 * p * p / 3.0 + p * p - 4.0 * t;
	coef2[1] =  2.0 * p / 27.0 * (36.0 * t - p *p) - q * q;


	//std::cout << b2 /27.0 << " " << (2 * b2 * b2 - 9 * c2)<< " " << d2 << std::endl;
	//std::cout << coef2[0] << " " << coef2[1] << std::endl;

	int nRootlambda = resolveRealPolynome3without2term(rootlambda, coef2);
	// il y a eu un changement de variable donc il faut d�caller les racines
	double offset = 2.0 * p / 3.0; // b/3a du poly du 2nd ordre

	
	if (nRootlambda == 1) { // une r�elle et 2 compl
		rootlambda[0] = signe * sqrt(rootlambda[0] - offset); // racine r�elle
		rootlambda[1] = rootlambda[1] - offset; // Partie relle de la racine 


		double terme = sqrt((rootlambda[1] + sqrt(rootlambda[1] * rootlambda[1] + rootlambda[2] * rootlambda[2])) / 2.0);
		
		root[0] = 0.5 * (rootlambda[0] + 2 * terme);
		root[1] = 0.5 * (rootlambda[0] - 2 * terme);

		for (int k = 0; k < 2; k++)
		{
			root[k] = root[k] - b / 4;
		}

		return 2;

	}
	else { // 3 r�elles
		for (int i = 0; i < 3; i++) {
			if (rootlambda[i] - offset < 0) {
				return 0; // que des racines n�gatives -> pas de racines r�elles
			}
			else {
				rootlambda[i] = signe * sqrt(rootlambda[i] - offset);
				
			}
		}
		
		root[0] = 0.5 * ( rootlambda[0] + rootlambda[1] + rootlambda[2]);
		root[1] = 0.5 * ( rootlambda[0] - rootlambda[1] - rootlambda[2]);
		root[2] = 0.5 * (-rootlambda[0] + rootlambda[1] - rootlambda[2]);
		root[3] = 0.5 * (-rootlambda[0] - rootlambda[1] + rootlambda[2]);

		for (int k = 0; k < 4; k++)
		{
			root[k] = root[k] - b / 4;
		}

		return 4;

	}
}


int resolveRealPolynome3without2termEigen(double* root, double* coef) {
	/*
	* return : the number of real root for the polynome x^3 + px + q = 0
	* root : is a array of size 3
	* coeff : is a array of size 2 (p and q of x^3 + px + q = 0)
	*/
	int nRoot = 0;

	Eigen::Vector4d coeff(coef[1], coef[0], 0 , 1); //double coef4[4] = { 1, -5, 3, 1 };
	Eigen::PolynomialSolver<double, Eigen::Dynamic> solver;
	solver.compute(coeff);
	const Eigen::PolynomialSolver<double, Eigen::Dynamic>::RootsType& r = solver.roots();

	//std::cout << r << std::endl;

	for (int k = 0; k < 3; k++) {
		if (r(k).imag() == 0) { // racine r�elle
			root[nRoot] = r(k).real();
			nRoot++;
		}
	}

	return nRoot;
}

int resvolveRealPolynome4without2termEigen(double* root, double* coef)
{
	/*
	* return : the number of real root for the polynome x^4 + bx^3 + dx + e = 0
	* root : is a array of size 4
	* coeff : is a array of size 3 (b,d,e)
	*/

	int nRoot = 0;
	Eigen::VectorXd coeff(5);
	coeff(0) = coef[2];
	coeff(1) = coef[1];
	coeff(2) = 0;
	coeff(3) = coef[0];
	coeff(4) = 1;

	Eigen::PolynomialSolver<double, Eigen::Dynamic> solver;
	solver.compute(coeff);
	const Eigen::PolynomialSolver<double, Eigen::Dynamic>::RootsType& r = solver.roots();

	for (int k = 0; k < 4; k++) {
		if (r(k).imag() == 0) { // racine r�elle
			root[nRoot] = r(k).real();
			nRoot++;
		}
	}

	return nRoot;

}


int resvolveRealPolynome4without2term(double* root, double* coef, bool Lagrange) {
	if (Lagrange) {
		return resvolveRealPolynome4without2termLagrange(root, coef);
	}
	else {
		return resvolveRealPolynome4without2term(root, coef);
	}
}


__host__ __device__ double findAntpoly3Neg(double p, double q) {
	/* 
		find one x where f(x) <0
		with f(x) = x^3 + px + q
	*/

	if (q > 0) {
		if (p > 0) {
			return -(q / p);
		}
		else {
			return -1.26*MAX(MAX(1 , -  p),  q);
		}
	}
	else {
		return 0;
	}


}
__host__ __device__ double findAntpoly3Pos(double p, double q) {

	/*
		find one x where f(x) >0
		with f(x) = x^3 + px + q
	*/

	if (q < 0) {
		if (p > 0) {
			return -(q / p);
		}
		else {
			return 1.26*MAX(MAX(1, -p), -q);
		}
	}
	else {
		return 0;
	}

}


__device__ int resolveRealPolynome3without2termGPU(double* root, double p, double q) {

	double Delta = -4.0 * p * p * p - 27.0 * q * q;
	if (Delta == 0) {
		root[0] = -3.0 * q / (2.0 * p);
		root[1] = -2.0 * root[0];
		return 2;
	}
	else if (Delta < 0) {
		double z0 =  cbrt((-q + sqrt(-Delta / 27.0)) / 2.0) + cbrt((-q - sqrt(-Delta / 27.0)) / 2.0);
		root[0] = z0; // b2 = z0
		
		double c2 = p + z0*z0;
		double delta2 = z0 * z0 - 4 * c2; // n�gatif normalement
		root[1] = -z0 / 2; // partie r�elle de la racine double
		root[2] = sqrt(-delta2) / 2;
		return 1;
	}
	else {

		for (int k = 0; k < 3; k++) {
			double r = (3.0 * q * sqrt(3.0)) / (2.0 * p * sqrt(-p));
			r = -1.0 * (r < -1.0) + 1.0 * (r > 1.0) + r * (r<1.0 && r>-1.0);
			root[k] = 2.0 * sqrt(-p / 3.0) * cos((acos(r) + 2.0 * k * PI) / 3.0);
		}

		return 3;
	}
}

/**/
__device__ int resvolveRealPolynome4without2termGPU(double* root, double b, double d, double e)
{
	/*
	* return : the number of real root for the polynome x^4 + bx^3 + dx + e = 0
	* root : is a array of size 4
	* coeff : is a array of size 3 (b,d,e)
	*/
	int nRoot = 0;

	if (b * b * b + 8.0 * d == 0) {

		// passage de p^4 + b p^3 + d p + e -> a p^4 + b p^2 + c = 0
		double B = -3.0 * b * b / 8.0;
		double C = -3.0 * b * b * b * b / 256.0 - b * d / 4.0 + e;

		double delta = B * B - 4.0 * C;
		//std::cout << "Delta " << delta;
		if (delta == 0) {
			double z = -B / 2.0;
			nRoot = 2;
			//std::cout << " z " << z << std::endl;
			root[0] = sqrt(z);
			root[1] = -sqrt(z);
			return nRoot;
		}
		else if (delta > 0) {
			double z1 = (-B + sqrt(delta)) / 2.0;
			double z2 = (-B - sqrt(delta)) / 2.0;
			//std::cout << " z1 " << z1 << " z2 " << z2 << std::endl;
			if (z1 >= 0) {
				root[0] = sqrt(z1);
				root[1] = -sqrt(z1);
				nRoot = 2;
			} if (z2 >= 0) {
				root[nRoot] = sqrt(z2);
				root[nRoot + 1] = -sqrt(z2);
				nRoot += 2;
			}
			return nRoot;
		}
	}


	double rootlambda[3];
	double coef2_0 = (2.0 * b * d - 8.0 * e) / 8.0;
	double coef2_1 = -(b * b * e + d * d) / 8.0;
	int nRootlambda = resolveRealPolynome3without2termGPU(rootlambda, coef2_0, coef2_1);

	for (int i = 0; i < nRootlambda; i++) {
		double lambda0 = rootlambda[i];
		double mu1 = 2.0 * lambda0 + (b * b) / 4.0;
		if (mu1 > 0) {
			double mu0 = sqrt(mu1);
			double DeltaP = -2.0 * lambda0 + 2.0 * (d - b * lambda0) / mu0 + b * mu0 + b * b / 2.0;
			double DeltaM = -2.0 * lambda0 - 2.0 * (d - b * lambda0) / mu0 - b * mu0 + b * b / 2.0;
			if (DeltaP >= 0) {
				root[nRoot] = (-mu0 + sqrt(DeltaP)) / 2.0 - b / 4.0;
				root[nRoot + 1] = (-mu0 - sqrt(DeltaP)) / 2.0 - b / 4.0;
				nRoot = nRoot + 2;
			}
			if (DeltaM >= 0) {
				root[nRoot] = (mu0 + sqrt(DeltaM)) / 2.0 - b / 4.0;
				root[nRoot + 1] = (mu0 - sqrt(DeltaM)) / 2.0 - b / 4.0;
				nRoot = nRoot + 2;
			}
			if (nRoot > 0) {
				return nRoot;
			}
		}
	}
	return nRoot;
}

__device__ int resvolveRealPolynome4without2termGPULagrange(double* root, double b, double d, double e) {

	int nRoot = 0;

	// il faut passer de b d e � p q t, c'est le coef devant z^3 qui doit etre nul pas celui devant z^2 !!!

	double p = -3.0 * b * b / (8.0);
	double q = d + (b * b * b) / 8.0;
	double t = e - b * d / 4.0 - 3 * b * b * b * b / 256;

	//t = e/a - b*d/(4*a^2) + c*b^2/(16*a^3) - 3*b^4/(256 * a^4);


	int signe = 1 - 2 * (q > 0);

	double rootlambda[3];
	double coef2_0 = -4.0 * p * p / 3.0 + p * p - 4.0 * t;
	double coef2_1 = 2.0 * p / 27.0 * (36.0 * t - p * p) - q * q;


	
	int nRootlambda = resolveRealPolynome3without2termGPU(rootlambda, coef2_0, coef2_1);
	// il y a eu un changement de variable donc il faut d�caller les racines
	double offset = 2.0 * p / 3.0; // b/3a du poly du 2nd ordre


	if (nRootlambda == 1) { // une r�elle et 2 compl
		rootlambda[0] = signe * sqrt(rootlambda[0] - offset); // racine r�elle
		rootlambda[1] = rootlambda[1] - offset; // Partie relle de la racine 


		double terme = sqrt((rootlambda[1] + sqrt(rootlambda[1] * rootlambda[1] + rootlambda[2] * rootlambda[2])) / 2.0);
	
		root[0] = 0.5 * (rootlambda[0] + 2 * terme);
		root[1] = 0.5 * (rootlambda[0] - 2 * terme);

		for (int k = 0; k < 2; k++)
		{
			root[k] = root[k] - b / 4;
		}

		return 2;

	}
	else { // 3 r�elles
		for (int i = 0; i < 3; i++) {
			if (rootlambda[i] - offset < 0) {
				return 0; // que des racines n�gatives -> pas de racines r�elles
			}
			else {
				rootlambda[i] = signe * sqrt(rootlambda[i] - offset);

			}
		}
		root[0] = 0.5 * (rootlambda[0] + rootlambda[1] + rootlambda[2]);
		root[1] = 0.5 * (rootlambda[0] - rootlambda[1] - rootlambda[2]);
		root[2] = 0.5 * (-rootlambda[0] + rootlambda[1] - rootlambda[2]);
		root[3] = 0.5 * (-rootlambda[0] - rootlambda[1] + rootlambda[2]);

		for (int k = 0; k < 4; k++)
		{
			root[k] = root[k] - b / 4;
		}

		return 4;
	}
}

__device__ int resvolveRealPolynome4without2termGPU(double* root, double b, double d, double e, bool Lagrange) {
	if (Lagrange) {
		return resvolveRealPolynome4without2termGPULagrange(root, b, d, e);
	}
	else {
		return resvolveRealPolynome4without2termGPU(root, b, d, e);
	}
}


__device__ void coefPolynome3From4to2coefGPU(double* coef4, double* coef2) {
	double a = coef4[0];

	double b = coef4[1];
	double c = coef4[2];
	double d = coef4[3];

	coef2[0] = (3.0 * a * c - b * b) / (3.0 * a * a);
	coef2[1] = (2.0 * b * b * b - 9.0 * a * b * c + 27.0 * a * a * d) / (27.0 * a * a * a);
}


__global__ void resolveSeveralRealPolynome3termGPU(double* nRoot, double* roots, double* coefs, int nPoly) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int step = blockDim.x * gridDim.x;

	for (int i = index; i < nPoly; i += step) {
		double coefsLocal2[2];
		double rootsLocal[3];
		double coefsLocal4[4];
		int nRootLocal = 0;
		for (int j = 0; j < 4; j++) {
			coefsLocal4[j] = coefs[j * nPoly + i];
		}
		if (coefsLocal4[0] == 0) { // polynone of degre 2
			if (coefsLocal4[1] == 0) { // polynom of degre 1
				if (coefsLocal4[2] != 0) { // no const
					nRootLocal = 1;
					rootsLocal[0] = -coefsLocal4[3] / coefsLocal4[2];
				}
				nRootLocal = 0;
			}
			else {
				// la flemme WIP
			}
		}
		else {
			
			
			coefPolynome3From4to2coefGPU(coefsLocal4, coefsLocal2);
			
			

			nRootLocal = resolveRealPolynome3without2termGPU(rootsLocal, coefsLocal2[0], coefsLocal2[1]);
			for (int k = 0; k < nRootLocal; k++) {
				rootsLocal[k] += -coefsLocal4[1] / (3 * coefsLocal4[0]);
			}
		}
		for (int j = 0; j < nRootLocal; j++) {
			roots[j * nPoly + i] = rootsLocal[j];
		}
		nRoot[i] = nRootLocal;

	}
}

__global__ void resolveSeveralRealPolynome4WO2termGPU(double* nRoot, double* roots, double* coefs, int nPoly) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int step = blockDim.x * gridDim.x;

	for (int i = index; i < nPoly; i += step) {
		double rootsLocal[4];
		double coefsLocal3[3];
		int nRootLocal = 0;
		for (int j = 0; j < 4; j++) {
			coefsLocal3[j] = coefs[j * nPoly + i];
		}
		
		nRootLocal = resvolveRealPolynome4without2termGPU(rootsLocal, coefsLocal3[0], coefsLocal3[2], coefsLocal3[3]);
		
		
		for (int j = 0; j < nRootLocal; j++) {
			roots[j * nPoly + i] = rootsLocal[j];
		}
		nRoot[i] = nRootLocal;

	}
}


__global__ void resolveSeveralRealPolynome4WO2termGPULagrange(double* nRoot, double* roots, double* coefs, int nPoly) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int step = blockDim.x * gridDim.x;

	for (int i = index; i < nPoly; i += step) {
		double rootsLocal[4];
		double coefsLocal3[3];
		int nRootLocal = 0;
		for (int j = 0; j < 4; j++) {
			coefsLocal3[j] = coefs[j * nPoly + i];
		}

		nRootLocal = resvolveRealPolynome4without2termGPULagrange(rootsLocal, coefsLocal3[0], coefsLocal3[2], coefsLocal3[3]);


		for (int j = 0; j < nRootLocal; j++) {
			roots[j * nPoly + i] = rootsLocal[j];
		}
		nRoot[i] = nRootLocal;

	}
}



__device__ int resolveRealPolynome3GPU(double* root, double b, double c, double d) {

	double x_i = 0;
	int nRoot = 0;
	int i = 0;
	double x_pre = 0;

	if (d != 0) {
		// solve Laguerre

		double eps = 2 * EPSNEWTON;

		while (i<ITERNEWTON && eps>EPSNEWTON)
		{
			//std::cout << i << " : " << x_i << std::endl;
			double p = F3(b, c, d, x_pre);
			double p2 = FPRIM3(b, c, x_pre);
			double p3 = FSECON3(b, x_pre);

			x_i = x_pre - (2 * p * p2) / (2 * p2 * p2 - p * p3);

			eps = (x_i - x_pre) * (x_i - x_pre);
			x_pre = x_i;
			i++;
		}
	}
	if (i == ITERNEWTON) {
		i = 0; // on r�essaie avec une autre init

		double p = (3 * c - b * b) / (3);
		double q = (2 * b * b * b - 9 * b * c + 27 * d) / (27);

		if (F3(0, p, q, 0) > 0) {
			x_pre = findAntpoly3Neg(p, q);
		}
		else {
			x_pre = findAntpoly3Pos(p, q);
		}
		double eps = 2 * EPSNEWTON;

		while (i<ITERNEWTON && eps>EPSNEWTON)
		{
			//std::cout << i << " : " << x_i << std::endl;
			double p1 = F3(0, p, q, x_pre);
			double p2 = FPRIM3(0, p, x_pre);
			double p3 = FSECON3(0, x_pre);

			x_i = x_pre - (2 * p1 * p2) / (2 * p2 * p2 - p1 * p3);

			eps = (x_i - x_pre) * (x_i - x_pre);
			x_pre = x_i;
			i++;
		}
		x_i = x_i - b / 3;

		if (i == ITERNEWTON) { // racine non trouv� ...
			double coefTemp[2];

			coefTemp[0] = p;
			coefTemp[1] = q;
			nRoot = resolveRealPolynome3without2termGPU(root, p, q);
			for (int k = 0; k < nRoot; k++) {
				root[k] = root[k] - b / 3;
			}

			return nRoot;
		}
	}
	root[nRoot] = x_i;
	nRoot++;

	// second degr�
	// x^2 + b x + c = 0 tel que  x^3 + s x^2 + px + q = (x-x_i)(x^2 + b x + c)
	double B = b + x_i;
	double C = c + x_i * B;


	double delta = B * B - 4 * C;

	if (delta == 0) {
		double z = -B / 2;
		root[nRoot] = z;
		nRoot++;
		return nRoot;
	}
	else if (delta > 0) {
		double z1 = (-B + sqrt(delta)) / 2;
		double z2 = (-B - sqrt(delta)) / 2;
		root[nRoot] = z1;
		nRoot++;
		root[nRoot] = z2;
		nRoot++;
		return nRoot;
	}
	
	return nRoot;

}/**/
__device__ int resvolveRealPolynome4GPU(double* root, double b, double c, double d, double e) {
	double x_i = 0;

	 
	if (e != 0) {
		 
		double x_pre = 0;

		int i = 0;
		double eps = 2 * EPSNEWTON;

		while (i<ITERNEWTON && eps>EPSNEWTON)
		{
			double p = F4(b, c, d, e, x_pre);
			double p2 = FPRIM4(b, c, d, x_pre);
			double p3 = FSECON4(b, c, x_pre);

			x_i = x_pre - (2 * p * p2) / (2 * p2 * p2 - p * p3);

			eps = (x_i - x_pre) * (x_i - x_pre);
			x_pre = x_i;
			i++;
		}
	}


	// troisi�me degr�
	 

	double B = b + x_i;
	double C = c + x_i * B;
	double D = d + x_i * C;

	/*double p = (3  * c - b * b) / (3);
	//double q = (2 * b * b * b - 9  * b * c + 27  * d) / (27);
	//coef3[0] = 0;
	//coef3[1] = p;
	//coef3[2] = q;
	std::cout << "Nouveau Polynome " << coef3[0] << " " << coef3[1] << " " << coef3[2] << std::endl;
	std::cout << "Nouveau Polynome ou " << b << " " << c << " " << d << std::endl;*/
 

	int nRoot = resolveRealPolynome3GPU(root, B, C, D);
	/*for (int i = 0; i < nRoot; i++) {
		root[i] = root[i] - b / 3;
	}*/
	root[nRoot] = x_i;
	nRoot++;


	return nRoot;

}



__global__ void resolveSeveralRealPolynome3GPU(double* nRoot, double* roots, double* coefs, int nPoly) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int step = blockDim.x * gridDim.x;

	for (int i = index; i < nPoly; i += step) {
		double coefsLocal3[3];
		double rootsLocal[3];
		double coefsLocal4[4];
		int nRootLocal = 0;
		for (int j = 0; j < 4; j++) {
			coefsLocal4[j] = coefs[j * nPoly + i];
		}
		coefsLocal3[0] = coefsLocal4[1] / coefsLocal4[0];
		coefsLocal3[1] = coefsLocal4[2] / coefsLocal4[0];
		coefsLocal3[2] = coefsLocal4[3] / coefsLocal4[0];

		nRootLocal = resolveRealPolynome3GPU(rootsLocal, coefsLocal3[0], coefsLocal3[1], coefsLocal3[2]);
		

		for (int j = 0; j < nRootLocal; j++) {
			roots[j * nPoly + i] = rootsLocal[j];
		}
		nRoot[i] = nRootLocal;
	}

}


__global__ void resolveSeveralRealPolynome4GPU(double* nRoot, double* roots, double* coefs, int nPoly) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int step = blockDim.x * gridDim.x;

	for (int i = index; i < nPoly; i += step) {
		double rootsLocal[4];
		double coefsLocal4[4];
		int nRootLocal = 0;
		for (int j = 0; j < 4; j++) {
			coefsLocal4[j] = coefs[j * nPoly + i];
		}

		nRootLocal = resvolveRealPolynome4GPU(rootsLocal, coefsLocal4[0], coefsLocal4[1], coefsLocal4[2], coefsLocal4[3]);


		for (int j = 0; j < nRootLocal; j++) {
			roots[j * nPoly + i] = rootsLocal[j];
		}
		nRoot[i] = nRootLocal;

	}
}




