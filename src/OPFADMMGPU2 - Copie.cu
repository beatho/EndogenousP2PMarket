#include "hip/hip_runtime.h"
#include "../head/OPFADMMGPU2.cuh"
#define MAX(X, Y) X * (X >= Y) + Y * (Y > X)
#define NMAXAGENTPERTHREAD 5

OPFADMMGPU2::OPFADMMGPU2() : MethodOPFGPU()
{
#if DEBUG_CONSTRUCTOR
	std::cout << " OPFADMMGPU2 Constructor" << std::endl;
#endif // DEBUG_CONSTRUCTOR
	_name = NAME;
	timePerBlock = MatrixCPU(1, 12, 0); // Fb0, Fb11abcd, FB12, Fb2, Fb3, Fb4, Fb5,FB6, Fb0'
	// si les sous ensemble ne sont pas accessible, tout est dans le premier.
	occurencePerBlock = MatrixCPU(1, 12, 0); //nb de fois utilisé pendant la simu
}

OPFADMMGPU2::OPFADMMGPU2(float rho) : MethodOPFGPU()
{
#if DEBUG_CONSTRUCTOR
	std::cout << "default OPFADMMGPU2 Constructor" << std::endl;
#endif // DEBUG_CONSTRUCTOR
	_name = NAME;
	_rho = rho;
	timePerBlock = MatrixCPU(1, 12, 0); // Fb0, Fb11, FB12, Fb2, Fb3, Fb4, Fb5,FB6, Fb0'
	// si les sous ensemble ne sont pas accessible, tout est dans le premier.
	occurencePerBlock = MatrixCPU(1, 12, 0); //nb de fois utilisé pendant la simu
}

OPFADMMGPU2::~OPFADMMGPU2()
{
}
void OPFADMMGPU2::setParam(float rho)
{
	_rho = rho;
}

bool OPFADMMGPU2::chekcase()
{
	if (_nBus != (_nLine + 1)) {
		std::cout << "wrong number of line " << _nLine << "against " << _nBus << std::endl;
		return false;
	}
	for (int i = 0; i < _nLine; i++) {
		if (CoresLineBus.get(i, 1) != (i + 1)) {
			std::cout << "wrong numerotation of line " << CoresLineBus.get(i, 1) << "against " << (i + 1) << std::endl;
			return false;
		}
		if (CoresLineBus.get(i, 0) > CoresLineBus.get(i, 1)) {
			std::cout << "wrong numeoration of bus " << CoresLineBus.get(i, 0) << "against " << CoresLineBus.get(i, 1) << std::endl;
			return false;
		}
	}
	if (ZsRe.getNLin() == 0  || ZsIm.getNLin() == 0) {
		std::cout << "matrice non defined, ZsRe, Zs Im, Yd" << std::endl;
		ZsRe.display();
		ZsIm.display();
		return false;
	}

	return true;
}

void OPFADMMGPU2::solve(Simparam* result, const Simparam& sim, const StudyCase& cas)
{
#ifdef DEBUG_SOLVE
	cas.display();
	sim.display(1);
#endif // DEBUG_SOLVE
	
	clock_t tall =clock();
#ifdef INSTRUMENTATION
	std::chrono::high_resolution_clock::time_point t1;
	std::chrono::high_resolution_clock::time_point t2;
#endif // INSTRUMENTATION

	// FB 0
	if (_id == 0) {
#ifdef INSTRUMENTATION
		t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION
		init(sim, cas);
#ifdef INSTRUMENTATION
		hipDeviceSynchronize();
		t2 = std::chrono::high_resolution_clock::now();
		timePerBlock.increment(0, 0, std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
		occurencePerBlock.increment(0, 0, 1);
#endif // INSTRUMENTATION
	}
	
	_iterG = sim.getIterG();
	int iterL = sim.getIterL();
	_stepG = sim.getStepG();
	int stepL = sim.getStepL();
	
	float epsG = sim.getEpsG();
	float epsL = sim.getEpsL();
	float rhoInit = sim.getRho();
	
	
	float fc = 0;
	float resG = 2 * epsG;
	float resL = 2 * epsL;
	_iterGlobal = 0;
	
	/*Chat.display(true);
	Bpt2.display(true);
	CoresSoloBusAgent.display();
	Cost1.display();
	Cost2.display();
	Pmin.display();
	Pmax.display();
	std::cout << "------" << std::endl;*/
	
	while ((_iterGlobal < _iterG) && (resG>epsG)) {
		
		
		/*std::cout << "--------" << std::endl;
		
		std::cout << " X " << std::endl;
		X.display(true);
		std::cout << " Q " << std::endl;
		Q.display(true);
		std::cout << " Y " << std::endl;
		Y.display(true);
		std::cout << " Mu " << std::endl;
		Mu.display(true);
		std::cout << " Chat " << std::endl;
		Chat.display(true);
		*/
		

#ifdef INSTRUMENTATION
		t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION
		// on pourrait faire en 2 appels pour mieu parraléliser
		// on pourrait serialiser la gestion des agents
		updateXOPFADMM <<<_nBus, _blockSizeSmall >> > (X._matrixGPU, Chat._matrixGPU, VoltageLimit._matrixGPU, _nAgentByBus._matrixGPU, nChild._matrixGPU, _indiceBusBegin._matrixGPU, _CoresChatBegin._matrixGPU, 
			_CoresAgentBusBegin._matrixGPU, _CoresAgentBus._matrixGPU, Cost1._matrixGPU, Cost2._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, _rho,  _nBus, _nAgent, Lagrange);

#ifdef INSTRUMENTATION
		hipDeviceSynchronize();
		t2 = std::chrono::high_resolution_clock::now();
		timePerBlock.increment(0, 1, std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
		t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION

		CommunicationX();

#ifdef INSTRUMENTATION
		hipDeviceSynchronize();
		t2 = std::chrono::high_resolution_clock::now();
		timePerBlock.increment(0, 6, std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
		t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION
		
		updateGlobalProb();
		updateMu();
#ifdef INSTRUMENTATION
		hipDeviceSynchronize();
		t2 = std::chrono::high_resolution_clock::now();
		timePerBlock.increment(0, 7, std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
		t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION
		
		updateChat();
		
#ifdef INSTRUMENTATION
		hipDeviceSynchronize();
		t2 = std::chrono::high_resolution_clock::now();
		timePerBlock.increment(0, 8, std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
#endif // INSTRUMENTATION
		// FB 4
		if (!(_iterGlobal % _stepG)) {
#ifdef INSTRUMENTATION
			t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION
			resG = updateRes(_iterGlobal / _stepG);
			//std::cout << _iterGlobal << " " << _iterLocal << " " << resL << " " << resF.get(0, _iterGlobal / _stepG) << " " << resF.get(1, _iterGlobal / _stepG) << std::endl;
			//resG = 1;
#ifdef INSTRUMENTATION
			hipDeviceSynchronize();
			t2 = std::chrono::high_resolution_clock::now();
			timePerBlock.increment(0, 9, std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
#endif // INSTRUMENTATION
		}
		//std::cout << iterGlobal << " " << _iterLocal << " " << resL << " " << resF.get(0, iterGlobal / stepG) << " " << resF.get(1, iterGlobal / stepG) << std::endl;

		_iterGlobal++;
	}
	//std::cout << iterGlobal << " " << _iterLocal << " " << resL << " " << resF.get(0, (iterGlobal - 1) / stepG) << " " << resF.get(1, (iterGlobal - 1) / stepG) << " " << resF.get(2, (iterGlobal - 1) / stepG) << std::endl;


#ifdef INSTRUMENTATION	
	occurencePerBlock.increment(0, 1, _iterGlobal);
	occurencePerBlock.increment(0, 6, _iterGlobal);
	occurencePerBlock.increment(0, 7, _iterGlobal);
	occurencePerBlock.increment(0, 8, _iterGlobal);
	occurencePerBlock.increment(0, 9, _iterGlobal / _stepG);

	t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION
	
	setPnFromX << < _nBus, _blockSizeSmall >> > (Pn._matrixGPU, X._matrixGPU, _indiceBusBegin._matrixGPU, _CoresAgentBus._matrixGPU, _nAgentByBus._matrixGPU, _CoresAgentBusBegin._matrixGPU, _nAgent);

	/*std::cout << "--------" << std::endl;
	std::cout << " X " << std::endl;
	X.display(true);
	std::cout << " Pn " << std::endl;
	Pn.display(true);
	*/
	
	fc = calcFc(&Cost1, &Cost2, &Pn, &tempN2);
	// FB 5
	
	result->setResF(&resF);
	result->setIter(_iterGlobal);
	MatrixCPU PnCPU;
	Pn.toMatCPU(PnCPU);

	result->setPn(&PnCPU);
	
	result->setFc(fc);

#ifdef INSTRUMENTATION
	hipDeviceSynchronize();
	t2 = std::chrono::high_resolution_clock::now();
	timePerBlock.increment(0, 10, std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
	occurencePerBlock.increment(0, 10, 1);

	result->setTimeBloc(&timePerBlock, &occurencePerBlock);
#endif // INSTRUMENTATION
	tall = clock() - tall;
	timeOPF = tall;

	result->setTime((float)tall / CLOCKS_PER_SEC);
	
}

void OPFADMMGPU2::updateP0(const StudyCase& cas)
{
	_id = _id + 1;
#ifdef INSTRUMENTATION
	std::chrono::high_resolution_clock::time_point t1 = std::chrono::high_resolution_clock::now();
#endif
	Pmin = cas.getPmin();
	Pmax = cas.getPmax();
	Cost2 = cas.getb();
	
	// pour essayer que cela marche
	Pn.add(&Pmin, &Pmax);
	Pn.divide(2);
	// remove loss agent
	Pn.set(0, 0, 0, 1);
	Pmin.set(0, 0, 0, 1);
	Pmax.set(0, 0, 0, 1);
	Pn.set(_nAgent, 0, 0, 1);
	Pmin.set(_nAgent, 0, 0, 1);
	Pmax.set(_nAgent, 0, 0, 1);

	ComputePFromAgentToBus();
	
	initPQAgent << < _nBus, _blockSize >> > (X._matrixGPU, _indiceBusBegin._matrixGPU, _CoresAgentBus._matrixGPU, _nAgentByBus._matrixGPU, _CoresAgentBusBegin._matrixGPU, Pn._matrixGPU, _nAgent);

	//_global__ void initDFSPQ(float* X, float* Pb, float* nChild, float* Childs, float* indiceBusBegin, float* indiceChildBegin, int nBus)
	initDFSPQ << <1, _nBus, _nBus* (8*sizeof(bool) + sizeof(int)) >> > (X._matrixGPU, Pb._matrixGPU, nChild._matrixGPU, Childs._matrixGPU, _indiceBusBegin._matrixGPU, _indiceChildBegin._matrixGPU, _nBus);
	communicateX << <_nBus, _blockSize >> > (X._matrixGPU, nChild._matrixGPU, Ancestor._matrixGPU, Childs._matrixGPU, _indiceBusBegin._matrixGPU, _indiceChildBegin._matrixGPU, _nAgentByBus._matrixGPU, _nBus);



	
	Y.set(&X);
#ifdef INSTRUMENTATION
	hipDeviceSynchronize();
	std::chrono::high_resolution_clock::time_point t2 = std::chrono::high_resolution_clock::now();
	timePerBlock.increment(0, 11, std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
	occurencePerBlock.increment(0, 11, 1);
	t1 = std::chrono::high_resolution_clock::now();
#endif

	updateChat();
#ifdef INSTRUMENTATION
	hipDeviceSynchronize();
	t2 = std::chrono::high_resolution_clock::now();
	timePerBlock.increment(0, 8, std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
	occurencePerBlock.increment(0, 8, 1);
	
#endif


}

void OPFADMMGPU2::init(const Simparam& sim, const StudyCase& cas)
{
	// intialisation des matrixs et variables 
	
	clock_t t = clock();
	//std::cout << "init " << std::endl;
	_rho = sim.getRho();
	
	if (_rhol == 0) {
		_rhol = _rho;
	}
	if (consensus) {
		std::cout << "pas coder pour update Q !!!" << std::endl;
		exit(-1);
	}
	
	const int iterG = sim.getIterG();
	const int stepG = sim.getStepG();
	
	_nAgent = cas.getNagent();
	
	_nBus = cas.getNBus();
	_nLine = cas.getNLine(true); // ne doit pas �tre r�duit ici !!!
	_sizeOPFTotal = 3 * _nLine + 5 * _nBus + 2 * (_nAgent - 1); // L = nChild.sum()
	_sizeChat = 4 * _nBus + 2 * (_nAgent - 1);
	_numBlocksN = ceil((_nAgent + _blockSize - 1) / _blockSize);
	_numBlocksM = ceil((_sizeOPFTotal + _blockSize - 1) / _blockSize);
	_numBlocksB = ceil((_nBus + _blockSize - 1) / _blockSize);

	
	
	// il faut remettre sur CPU ce qu'il faut !!!
	if (tempL.getPos()) {
		tempL.transferCPU();
		_CoresChatBegin.transferCPU();
		_indiceBusBegin.transferCPU();
		Ancestor.transferCPU();
		PosChild.transferCPU();

		_indiceChildBegin.transferCPU();
		Childs.transferCPU();
	}


	tempL = MatrixGPU(_nLine, 1);


	//std::cout << _nAgent << " " << _nBus << " " << _nLine << std::endl;
	
	nChildCPU = MatrixCPU(_nBus, 1);
	CoresLineBus = cas.getCoresLineBus(true);
	_CoresBusAgent = cas.getCoresBusAgentLin(); // Cores[n] = b
	Ancestor = MatrixGPU(_nBus, 1, 0); // A_i = bus ant�c�dent de i
	Ancestor.set(0, 0, -1); // the slack bus has no ancestor
	ZsRe = MatrixGPU(cas.getZsRe());
	ZsIm = MatrixGPU(cas.getZsImag());
	ZsNorm = MatrixGPU(_nLine, 1);
	ZsNorm.multiplyT(&ZsRe, &ZsRe);
	tempL.multiplyT(&ZsIm, &ZsIm);
	ZsNorm.add(&tempL);
	tempL.transferGPU();

	if (!chekcase()) {
		throw std::invalid_argument("not a radial case");
	}

	for (int lold = 0; lold < _nLine; lold++) {
		int l = lold + 1;
		int busTo = l ;
		int busFrom = CoresLineBus.get(lold, 0);
		Ancestor.set(busTo, 0, busFrom);
		nChildCPU.set(busFrom, 0, nChildCPU.get(busFrom, 0) + 1); // pas parallelisable -> reduction chelou
	}
	
	nChild = MatrixGPU(nChildCPU, 1);

	
	_rhoInv = 1 / _rho;
	resF = MatrixCPU(3, (iterG / stepG) + 1, 0);

	
	MatrixGPU lowerBound(cas.getLowerBound(), 1); //voltage angle, voltage, line...
	MatrixGPU upperBound(cas.getUpperBound(), 1); //voltage angle, voltage, line...
	

	//std::cout << " local resolution " << std::endl;
	// local resolution
	tempN2 = MatrixGPU(2 * _nAgent, 1, 0, 1);
	tempB2 = MatrixGPU(2 * _nBus, 1, 0, 1);
	CoresSoloBusAgent = MatrixGPU(_nBus, 1, -1, 1);
	Pn = MatrixGPU(sim.getPn(), 1);
	Pmin = MatrixGPU(cas.getPmin(), 1);
	Pmax = MatrixGPU(cas.getPmax(), 1);
	
	Pb = MatrixGPU(2 * _nBus, 1, 0, 1);
	Pbmin = MatrixGPU(2 * _nBus, 1, 0, 1);
	Pbmax = MatrixGPU(2 * _nBus, 1, 0, 1);

	Cost1 = MatrixGPU(cas.geta(), 1);
	Cost2 = MatrixGPU(cas.getb(), 1);

	Pn.preallocateReduction();
	if (Pn.max2() < 0.00001) {
		Pn.add(&Pmin, &Pmax);
		Pn.divide(2);
	}
	_CoresAgentBus = MatrixGPU(cas.getCoresAgentBusLin(), 1);
	_CoresAgentBusBegin = MatrixGPU(cas.getCoresAgentBusLinBegin(), 1);
	_nAgentByBus = MatrixGPU(cas.getNagentByBus(), 1);
	_nAgentByBusCPU = cas.getNagentByBus();
	// remove the grid agent


	Pn.set(0, 0, 0, 1);
	Pmin.set(0, 0, 0, 1);
	Pmax.set(0, 0, 0, 1);
	Pn.set(_nAgent, 0, 0, 1);
	Pmin.set(_nAgent, 0, 0, 1);
	Pmax.set(_nAgent, 0, 0, 1);

	_nAgentByBusCPU.increment(0, 0, -1);
	_nAgentOn0 = _nAgentByBusCPU.get(0, 0);
	//std::cout << "remove loss agent" << std::endl;
	removeLossAgent << <1, 1 >> > (_nAgentByBus._matrixGPU, _CoresAgentBusBegin._matrixGPU);


	ComputePFromAgentToBus();


	
	//std::cout << " creation " << std::endl;
	X = MatrixGPU(_sizeOPFTotal, 1, 0, 1); // Changement d'ordre !!!!!!!!!!!!
	Ypre = MatrixGPU(_sizeOPFTotal, 1, 0, 1); // (Pi, Qi, li, vi, pn..., qn..., vai, Pci ..., Qci... , lci...) !!!!!
	Y = MatrixGPU(_sizeOPFTotal, 1, 0, 1);
	Y.preallocateReduction();
	//YTrans = MatrixGPU(_sizeOPFTotal, 1, 0, 1);
	Mu = MatrixGPU(_sizeOPFTotal, 1, 0, 1);
	
	tempN1 = MatrixGPU(_nAgent, 1, 0, 1);
	tempNN = MatrixGPU(_nAgent, _nAgent, 0, 1);
	//tempM1 = new MatrixGPU[_nAgent];
	tempM = MatrixGPU(_sizeOPFTotal, 1, 0, 1);
	
	sizeOPFADMMGPU2 = MatrixGPU(_nBus, 1, 0, 1);
	sizeOPFADMMGPU2.preallocateReduction();
	sizeOPFADMMGPU2Big = MatrixGPU(_sizeOPFTotal, 1, 0, 1);

	_indiceBusBegin = MatrixGPU(_nBus, 1);
	_indiceBusBeginBig = MatrixGPU(_sizeOPFTotal, 1, 0, 1);
	_CoresChatBegin = MatrixGPU(_nBus, 1);
	
	int debut = 0;
	int debutChat = 0;
	for (int i = 0; i < _nBus; i++) {
		int m = nChildCPU.get(i, 0);
		int nB = _nAgentByBusCPU.get(i, 0);
		_indiceBusBegin.set(i, 0, debut);
		_CoresChatBegin.set(i, 0, debutChat);
		int sizeA = m * 3 + 5 + 2 * nB;
		debut += sizeA;
		debutChat += (4 + 2 * nB);
	}
	//_CoresChatBegin.display();


	_CoresChatBegin.transferGPU();
	_indiceBusBegin.transferGPU();
	defineSizeBig <<<_nBus, _blockSize >> > (sizeOPFADMMGPU2Big._matrixGPU, nChild._matrixGPU, _indiceBusBegin._matrixGPU, sizeOPFADMMGPU2._matrixGPU, _indiceBusBeginBig._matrixGPU, _nAgentByBus._matrixGPU);
	
	//sizeOPFADMMGPU2.display(true);


	_sizeOPFMax = sizeOPFADMMGPU2.max2();
	Hinv = MatrixGPU(_sizeOPFTotal, _sizeOPFMax, 0, 1);
	Q = MatrixGPU(_sizeOPFTotal, 1, 0, 1);
	
	Childs = MatrixGPU(_nLine, 1);
	PosChild = MatrixGPU(_nBus, 1, -1);

	Chat = MatrixGPU( _sizeChat, 1, 0, 1);
	VoltageLimit = MatrixGPU(2, _nBus, 0, 1); // min, max
	VoltageLimitReal = MatrixGPU(2, _nBus, 0, 1); // min, max
	
	
	_indiceChildBegin = MatrixGPU(_nLine, 1);
	//int sizeOPF2 = 1 * nChild.get(i, 0) + 9;
	
	
	MatrixCPU nChildTemp(_nBus, 1, 0);
	//lowerBound.display(true);
	//upperBound.display(true);
	initVoltageBound <<< _numBlocksB, _blockSize >> > (VoltageLimitReal._matrixGPU, VoltageLimit._matrixGPU, lowerBound._matrixGPU, upperBound._matrixGPU, nChild._matrixGPU, _nBus);

	//nChild.display(true);
	//VoltageLimit.display(true);
	//VoltageLimitReal.display(true);
	//

	//nChild.display();
	
	int debutChild = 0;
	for (int i = 0; i < _nBus; i++) {
		if (i > 0) {
			_indiceChildBegin.set(i - 1, 0, debutChild);
		
			int Ai = Ancestor.get(i, 0);
			Childs.set(_indiceChildBegin.get(Ai, 0) + nChildTemp.get(Ai, 0), 0, i);
			PosChild.set(i, 0, nChildTemp.get(Ai, 0));
			nChildTemp.increment(Ai, 0, 1);
			debutChild += nChildCPU.get(i - 1, 0);
		}
		
	}
	/*Childs.display();
	Ancestor.display();
	nChildCPU.display();
	PosChild.display();
	std::cout << " _indiceChildBegin " << std::endl;
	_indiceChildBegin.display(true);*/
	Ancestor.transferGPU();
	PosChild.transferGPU();
	debut = 0;
	//std::cout << " Hinv " << std::endl;
	for (int i = 0; i < _nBus; i++) {
		// (Pi, Qi, li, vi, pn..., qn..., vai, Pci ..., Qci... , lci...) !!!!!
		int m = nChildCPU.get(i, 0);
		int nB = _nAgentByBusCPU.get(i, 0);
		int sizeA = nChildCPU.get(i,0) * 3 + 5 + 2 * nB;
		MatrixCPU A(2 + 1 * (i > 0), sizeA);
		
		if (i > 0) {
			A.set(2, 0, 2 * ZsRe.get(i - 1, 0));
			A.set(2, 1, 2 * ZsIm.get(i - 1, 0));
			A.set(2, 2, -ZsNorm.get(i - 1, 0));
			A.set(2, 3, -1);
			A.set(2, 4 + 2 * nB, 1); // vai
			A.set(0, 0, -1);
			A.set(1, 1, -1);
		}
		for (int In = 0; In < nB; In++) {
			A.set(0, 4 + In, 1);
			A.set(1, 4 + nB + In, 1);
		}

		for (int j = 0; j < m; j++) {
			int c = Childs.get(_indiceChildBegin.get(i, 0) + j, 0);
			A.set(0, 5 + 2 * nB + j, 1); // Pci
			A.set(1, 5 + 2 * nB + m + j, 1); // Qci
			A.set(0, 5 + 2 * nB + 2 * m + j, -ZsRe.get(c - 1, 0)); // -R l
			A.set(1, 5 + 2 * nB + 2 * m + j, -ZsIm.get(c - 1, 0)); // -X l
		}
		
		//A.display();
		
		MatrixCPU temp33(2 + 1 * (i > 0), 2 + 1 * (i > 0));
		MatrixCPU temp3M(2 + 1 * (i > 0), sizeA);
		MatrixCPU tempMM(sizeA, sizeA);
		MatrixCPU tempMMbis(sizeA, sizeA);

		temp33.multiplyTrans(&A, &A);
		temp33.invertEigen(&temp33);
		temp3M.MultiplyMatMat(&temp33, &A);
		tempMM.multiplyTrans(&A, &temp3M, 0);

		tempMMbis.setEyes(-1);
		tempMMbis.add(&tempMM);
		MatrixGPU tempMMGPU = MatrixGPU(tempMMbis, 1);
		
		Hinv.setBloc(debut, debut + sizeA, 0, sizeA, &tempMMGPU);
		debut += sizeA;
	}
	Hinv.divide(_rho);
	//Hinv.display(true);
	_indiceChildBegin.transferGPU();
	Childs.transferGPU();
	//std::cout << " Childs " << std::endl;
	//Childs.display(true);
    //std::cout << " init valeur " << std::endl;
	
	initPQAgentV <<< _nBus, _blockSizeSmall >> > (X._matrixGPU, _indiceBusBegin._matrixGPU, _CoresAgentBus._matrixGPU, _nAgentByBus._matrixGPU, _CoresAgentBusBegin._matrixGPU, Pn._matrixGPU, _nAgent);
	
	
	/*std::cout << " X " << std::endl;
	X.display(true);
	std::cout << " _indiceBusBegin " << std::endl;
	_indiceBusBegin.display(true);
	std::cout << " _indiceChildBegin " << std::endl;
	_indiceChildBegin.display(true);
	std::cout << " Childs " << std::endl;
	Childs.display(true);
	std::cout << " nChild " << std::endl;
	nChild.display(true);
	std::cout << " posChild " << std::endl;
	PosChild.display(true);*/

	initDFSPQ << <1, _nBus, _nBus*(8*sizeof(bool) + sizeof(int)) >> > (X._matrixGPU, Pb._matrixGPU, nChild._matrixGPU, Childs._matrixGPU, _indiceBusBegin._matrixGPU, _indiceChildBegin._matrixGPU, _nBus);
	
	

	
	////CHECK_LAST_CUDA_ERROR();
	
	communicateX << <_nBus, _blockSize >> > (X._matrixGPU, nChild._matrixGPU, Ancestor._matrixGPU, Childs._matrixGPU, _indiceBusBegin._matrixGPU, _indiceChildBegin._matrixGPU, _nAgentByBus._matrixGPU, _nBus);

	
	Y.set(&X);
	/*std::cout << " X " << std::endl;
	X.display(true);
	std::cout << " Y " << std::endl;
	Y.display(true);
	std::cout << " Q " << std::endl;
	Q.display(true);
	std::cout << " Mu "<< std::endl;
	Mu.display(true);*/

	updateChat();
	/*std::cout << " Chat " << std::endl;
	Chat.display(true);*/
	//std::cout << " Bpt2 " << std::endl;
	//Bpt2.display(true);
	/*std::cout << " Nagent " << std::endl;
	_nAgentByBus.display(true);
	std::cout << " Bus Agent : agent->bus " << std::endl;
	_CoresBusAgent.display(true);
	std::cout << " Agent bus : bus->agent " << std::endl;
	_CoresAgentBus.display(true);
	std::cout << " Agent bus begin : bus->agent " << std::endl;
	_CoresAgentBusBegin.display(true);
	
	std::cout << "Apt2 " << std::endl;
	Apt2.display(true);
	std::cout << " Pmin " << std::endl;
	Pmin.display(true);
	std::cout << " Pma " << std::endl;
	Pmax.display(true);
	std::cout << " CoresSoloBusAgent " << std::endl;
	CoresSoloBusAgent.display(true);*/


	//std::cout << "fin init temps : " << (float)(clock() - t) / CLOCKS_PER_SEC << std::endl;
	//std::cout << "---------------------------------------------------------------------------------------" << std::endl;
}

void OPFADMMGPU2::solveConsensus(float eps, MatrixCPU* PSO)
{
	throw std::invalid_argument("WIP !!");
}

void OPFADMMGPU2::initConsensus(const Simparam& sim, const StudyCase& cas, float rhoSO)
{
	throw std::invalid_argument("WIP !!");
}

void OPFADMMGPU2::updateConsensus(MatrixCPU* Pmarket)
{
	throw std::invalid_argument("WIP !!");
}

void OPFADMMGPU2::solveConsensus(float eps, MatrixGPU* PSO)
{
	throw std::invalid_argument("WIP !!");
}

void OPFADMMGPU2::updateConsensus(MatrixGPU* Pmarket)
{
	throw std::invalid_argument("WIP !!");
}



void OPFADMMGPU2::updateGlobalProb() {
	
	Ypre.swap(&Y);
	int numBlock = _sizeOPFTotal;
	switch (_blockSize) {
	case 512:
		updateY<512> << <numBlock, _blockSize >> > (Y._matrixGPU, Hinv._matrixGPU, Q._matrixGPU, sizeOPFADMMGPU2Big._matrixGPU, _indiceBusBeginBig._matrixGPU, _sizeOPFMax);
		break;
	case 256:
		updateY<256> << <numBlock, _blockSize >> > (Y._matrixGPU, Hinv._matrixGPU, Q._matrixGPU, sizeOPFADMMGPU2Big._matrixGPU, _indiceBusBeginBig._matrixGPU, _sizeOPFMax);
		break;
	case 128:
		updateY<128> << <numBlock, _blockSize >> > (Y._matrixGPU, Hinv._matrixGPU, Q._matrixGPU, sizeOPFADMMGPU2Big._matrixGPU, _indiceBusBeginBig._matrixGPU, _sizeOPFMax);
		break;
	case 64:
		updateY< 64> << <numBlock, _blockSize >> > (Y._matrixGPU, Hinv._matrixGPU, Q._matrixGPU, sizeOPFADMMGPU2Big._matrixGPU, _indiceBusBeginBig._matrixGPU, _sizeOPFMax);
		break;
	case 32:
		updateY< 32> << <numBlock, _blockSize >> > (Y._matrixGPU, Hinv._matrixGPU, Q._matrixGPU, sizeOPFADMMGPU2Big._matrixGPU, _indiceBusBeginBig._matrixGPU, _sizeOPFMax);
		break;
	case 16:
		updateY< 16> << <numBlock, _blockSize >> > (Y._matrixGPU, Hinv._matrixGPU, Q._matrixGPU, sizeOPFADMMGPU2Big._matrixGPU, _indiceBusBeginBig._matrixGPU, _sizeOPFMax);
		break;
	case  8:
		updateY<  8> << <numBlock, _blockSize >> > (Y._matrixGPU, Hinv._matrixGPU, Q._matrixGPU, sizeOPFADMMGPU2Big._matrixGPU, _indiceBusBeginBig._matrixGPU, _sizeOPFMax);
		break;
	case  4:
		updateY<  4> << <numBlock, _blockSize >> > (Y._matrixGPU, Hinv._matrixGPU, Q._matrixGPU, sizeOPFADMMGPU2Big._matrixGPU, _indiceBusBeginBig._matrixGPU, _sizeOPFMax);
		break;
	case  2:
		updateY<  2> << <numBlock, _blockSize >> > (Y._matrixGPU, Hinv._matrixGPU, Q._matrixGPU, sizeOPFADMMGPU2Big._matrixGPU, _indiceBusBeginBig._matrixGPU, _sizeOPFMax);
		break;
	case  1:
		updateY<  1> << <numBlock, _blockSize >> > (Y._matrixGPU, Hinv._matrixGPU, Q._matrixGPU, sizeOPFADMMGPU2Big._matrixGPU, _indiceBusBeginBig._matrixGPU, _sizeOPFMax);
		break;
	}
	
	Y.set(3, 0, 1, 1);
	Y.set(4 + 2 * _nAgentOn0, 0, 1, 1);

}



void OPFADMMGPU2::updateMu()
{
	updateMUGPU << <_numBlocksM, _blockSize >> > (Mu._matrixGPU, Y._matrixGPU, X._matrixGPU, _rho, _sizeOPFTotal);	
}


float OPFADMMGPU2::getPLoss()
{
	
	return Pn.sum(0, _nAgent);
}

float OPFADMMGPU2::getQLoss()
{
	return Pn.sum(_nAgent, 2 * _nAgent);
}

void OPFADMMGPU2::ComputePFromAgentToBus()
{
	int numBlock = _nBus;
	switch (_blockSize) {
	case 512:
		ComputePFromAgentToBusGPU<512> << <numBlock, _blockSizeSmall >> > (Pb._matrixGPU, Pbmin._matrixGPU, Pbmax._matrixGPU, CoresSoloBusAgent._matrixGPU, Pn._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, _CoresAgentBus._matrixGPU, _nAgentByBus._matrixGPU, _CoresAgentBusBegin._matrixGPU, _nAgent, _nBus);
		break;
	case 256:
		ComputePFromAgentToBusGPU<256> << <numBlock, _blockSizeSmall >>>  (Pb._matrixGPU, Pbmin._matrixGPU, Pbmax._matrixGPU, CoresSoloBusAgent._matrixGPU, Pn._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, _CoresAgentBus._matrixGPU, _nAgentByBus._matrixGPU, _CoresAgentBusBegin._matrixGPU, _nAgent, _nBus);
		break;
	case 128:
		ComputePFromAgentToBusGPU<128> << <numBlock, _blockSizeSmall >>> (Pb._matrixGPU, Pbmin._matrixGPU, Pbmax._matrixGPU, CoresSoloBusAgent._matrixGPU, Pn._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, _CoresAgentBus._matrixGPU, _nAgentByBus._matrixGPU, _CoresAgentBusBegin._matrixGPU, _nAgent, _nBus);
		break;
	case 64:
		ComputePFromAgentToBusGPU< 64> << <numBlock, _blockSizeSmall >>> (Pb._matrixGPU, Pbmin._matrixGPU, Pbmax._matrixGPU, CoresSoloBusAgent._matrixGPU, Pn._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, _CoresAgentBus._matrixGPU, _nAgentByBus._matrixGPU, _CoresAgentBusBegin._matrixGPU, _nAgent, _nBus);
		break;
	case 32:
		ComputePFromAgentToBusGPU< 32> << <numBlock, _blockSizeSmall >>> (Pb._matrixGPU, Pbmin._matrixGPU, Pbmax._matrixGPU, CoresSoloBusAgent._matrixGPU, Pn._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, _CoresAgentBus._matrixGPU, _nAgentByBus._matrixGPU, _CoresAgentBusBegin._matrixGPU, _nAgent, _nBus);
		break;
	case 16:
		ComputePFromAgentToBusGPU< 16> << <numBlock, _blockSizeSmall >>> (Pb._matrixGPU, Pbmin._matrixGPU, Pbmax._matrixGPU, CoresSoloBusAgent._matrixGPU, Pn._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, _CoresAgentBus._matrixGPU, _nAgentByBus._matrixGPU, _CoresAgentBusBegin._matrixGPU, _nAgent, _nBus);
		break;
	case  8:
		ComputePFromAgentToBusGPU<  8> << <numBlock, _blockSizeSmall >>> (Pb._matrixGPU, Pbmin._matrixGPU, Pbmax._matrixGPU, CoresSoloBusAgent._matrixGPU, Pn._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, _CoresAgentBus._matrixGPU, _nAgentByBus._matrixGPU, _CoresAgentBusBegin._matrixGPU, _nAgent, _nBus);
		break;
	case  4:
		ComputePFromAgentToBusGPU<  4> << <numBlock, _blockSizeSmall >>> (Pb._matrixGPU, Pbmin._matrixGPU, Pbmax._matrixGPU, CoresSoloBusAgent._matrixGPU, Pn._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, _CoresAgentBus._matrixGPU, _nAgentByBus._matrixGPU, _CoresAgentBusBegin._matrixGPU, _nAgent, _nBus);
		break;
	case  2:
		ComputePFromAgentToBusGPU<  2> << <numBlock, _blockSizeSmall >>> (Pb._matrixGPU, Pbmin._matrixGPU, Pbmax._matrixGPU, CoresSoloBusAgent._matrixGPU, Pn._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, _CoresAgentBus._matrixGPU, _nAgentByBus._matrixGPU, _CoresAgentBusBegin._matrixGPU, _nAgent, _nBus);
		break;
	case  1:
		ComputePFromAgentToBusGPU<  1> << <numBlock, _blockSizeSmall >>> (Pb._matrixGPU, Pbmin._matrixGPU, Pbmax._matrixGPU, CoresSoloBusAgent._matrixGPU, Pn._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, _CoresAgentBus._matrixGPU, _nAgentByBus._matrixGPU, _CoresAgentBusBegin._matrixGPU, _nAgent, _nBus);
		break;
	}
}

void OPFADMMGPU2::updateChat()
{
	int numBlock = _nBus;
	switch (_blockSizeSmall) {
	case 512:
		updateChatGPU2<512> << <numBlock, _blockSizeSmall >> > (Chat._matrixGPU, Y._matrixGPU, Mu._matrixGPU, nChild._matrixGPU, Ancestor._matrixGPU, PosChild._matrixGPU, Childs._matrixGPU, _indiceBusBegin._matrixGPU, _indiceChildBegin._matrixGPU, _CoresChatBegin._matrixGPU, _nAgentByBus._matrixGPU, _rho, _nBus);
		break;
	case 256:
		updateChatGPU2<256> << <numBlock, _blockSizeSmall >> > (Chat._matrixGPU, Y._matrixGPU, Mu._matrixGPU, nChild._matrixGPU, Ancestor._matrixGPU, PosChild._matrixGPU, Childs._matrixGPU, _indiceBusBegin._matrixGPU, _indiceChildBegin._matrixGPU, _CoresChatBegin._matrixGPU, _nAgentByBus._matrixGPU, _rho, _nBus);
		break;
	case 128:
		updateChatGPU2<128> << <numBlock, _blockSizeSmall >> > (Chat._matrixGPU, Y._matrixGPU, Mu._matrixGPU, nChild._matrixGPU, Ancestor._matrixGPU, PosChild._matrixGPU, Childs._matrixGPU, _indiceBusBegin._matrixGPU, _indiceChildBegin._matrixGPU, _CoresChatBegin._matrixGPU, _nAgentByBus._matrixGPU, _rho, _nBus);
		break;
	case 64:
		updateChatGPU2< 64> << <numBlock, _blockSizeSmall >> > (Chat._matrixGPU, Y._matrixGPU, Mu._matrixGPU, nChild._matrixGPU, Ancestor._matrixGPU, PosChild._matrixGPU, Childs._matrixGPU, _indiceBusBegin._matrixGPU, _indiceChildBegin._matrixGPU, _CoresChatBegin._matrixGPU, _nAgentByBus._matrixGPU, _rho, _nBus);
		break;
	case 32:
		updateChatGPU2< 32> << <numBlock, _blockSizeSmall >> > (Chat._matrixGPU, Y._matrixGPU, Mu._matrixGPU, nChild._matrixGPU, Ancestor._matrixGPU, PosChild._matrixGPU, Childs._matrixGPU, _indiceBusBegin._matrixGPU, _indiceChildBegin._matrixGPU, _CoresChatBegin._matrixGPU, _nAgentByBus._matrixGPU, _rho, _nBus);
		break;
	case 16:
		updateChatGPU2< 16> << <numBlock, _blockSizeSmall >> > (Chat._matrixGPU, Y._matrixGPU, Mu._matrixGPU, nChild._matrixGPU, Ancestor._matrixGPU, PosChild._matrixGPU, Childs._matrixGPU, _indiceBusBegin._matrixGPU, _indiceChildBegin._matrixGPU, _CoresChatBegin._matrixGPU, _nAgentByBus._matrixGPU, _rho, _nBus);
		break;
	case  8:
		updateChatGPU2<  8> << <numBlock, _blockSizeSmall >> > (Chat._matrixGPU, Y._matrixGPU, Mu._matrixGPU, nChild._matrixGPU, Ancestor._matrixGPU, PosChild._matrixGPU, Childs._matrixGPU, _indiceBusBegin._matrixGPU, _indiceChildBegin._matrixGPU, _CoresChatBegin._matrixGPU, _nAgentByBus._matrixGPU, _rho, _nBus);
		break;
	case  4:
		updateChatGPU2<  4> << <numBlock, _blockSizeSmall >> > (Chat._matrixGPU, Y._matrixGPU, Mu._matrixGPU, nChild._matrixGPU, Ancestor._matrixGPU, PosChild._matrixGPU, Childs._matrixGPU, _indiceBusBegin._matrixGPU, _indiceChildBegin._matrixGPU, _CoresChatBegin._matrixGPU, _nAgentByBus._matrixGPU, _rho, _nBus);
		break;
	case  2:
		updateChatGPU2<  2> << <numBlock, _blockSizeSmall >> > (Chat._matrixGPU, Y._matrixGPU, Mu._matrixGPU, nChild._matrixGPU, Ancestor._matrixGPU, PosChild._matrixGPU, Childs._matrixGPU, _indiceBusBegin._matrixGPU, _indiceChildBegin._matrixGPU, _CoresChatBegin._matrixGPU, _nAgentByBus._matrixGPU, _rho, _nBus);
		break;
	case  1:
		updateChatGPU2<  1> << <numBlock, _blockSizeSmall >> > (Chat._matrixGPU, Y._matrixGPU, Mu._matrixGPU, nChild._matrixGPU, Ancestor._matrixGPU, PosChild._matrixGPU, Childs._matrixGPU, _indiceBusBegin._matrixGPU, _indiceChildBegin._matrixGPU, _CoresChatBegin._matrixGPU, _nAgentByBus._matrixGPU, _rho, _nBus);
		break;
	}
}

void OPFADMMGPU2::CommunicationX()
{
	 // X = { Pi, Qi, li, vi, (pn ...), qn..., vAi,  Pci ... , Qci ... , lci ... for all child Ci }
	
	communicateX << <_nBus, _blockSize >> > (X._matrixGPU, nChild._matrixGPU, Ancestor._matrixGPU, Childs._matrixGPU, _indiceBusBegin._matrixGPU, _indiceChildBegin._matrixGPU, _nAgentByBus._matrixGPU, _nBus);

	// Y = { Pi, Qi, vi, li, vAi, (pn ...), qn...,  Pci ... , Qci ... , lci ... for all child Ci }

	updateQ << <_numBlocksM, _blockSize >> > (Q._matrixGPU, X._matrixGPU, Mu._matrixGPU, _rho, _sizeOPFTotal);
}



float OPFADMMGPU2::updateRes(int indice) 
{
	
	float resS = _rho * Y.max2(&Ypre);
	float resR = Y.max2(&X);
	float resV = 0;
	
	resF.set(0, indice, resR);
	resF.set(1, indice, resS);
	resF.set(2, indice, resV);

	/*std::cout << resS << " " << resR << std::endl;
	std::cout << " Y " << std::endl;
	Y.display(true);
	std::cout << " X " << std::endl;
	X.display(true);*/


	if (resR > _mu * resS) {
		_rho = _tau * _rho;
		
		Hinv.divide(_tau);
		//std::cout << _iterGlobal << "rho augmente " << _rho << std::endl;
	}
	else if (resS > _mu * resR) {// rho = rho / tau_inc;
		_rho = _rho / _tau;
		Hinv.multiply(_tau);
		
		//std::cout << _iterGlobal << "rho diminue " << _rho << std::endl;
	}/**/


	return MAX(MAX(resV, resS), resR);
}

int OPFADMMGPU2::feasiblePoint()
{
	bool mustTrans = false;
	if (X.getPos()) {
		X.transferCPU();
		_indiceBusBegin.transferCPU();
		mustTrans = true;
	}
	// X  (Pi, Qi, li, vi, pn..., qn..., vai, Pci ..., Qci... , lci...) !!!!!

	MatrixCPU test(_nBus, 1, -1);
	int counter = 0;
	for (int bus = 0; bus < _nBus; bus++) {
		int begin = _indiceBusBegin.get(bus, 0);
		float Si = X.get(begin, 0) * X.get(begin, 0) + X.get(begin + 1, 0) * X.get(begin + 1, 0);
		float li = X.get(begin + 2, 0);
		float vi = X.get(begin + 3, 0);
		float err = Si - li * vi;
		test.set(bus, 0, err);
		if (abs(err) > 0.0001) {
			counter++;
		}
	}
	//std::cout << " erreur sur la relaXation " << test.max2() << " " << counter << std::endl;
	//test.display();

	if (mustTrans) {
		X.transferGPU();
		_indiceBusBegin.transferGPU();
	}
	resF.set(2, (_iterGlobal - 1) / _stepG, test.max2());
	return counter;
}




void OPFADMMGPU2::display() {

	std::cout.precision(3);

	X.transferCPU();
	Y.transferCPU();
	Mu.transferCPU();
	Pn.transferCPU();
	_indiceBusBegin.transferCPU();
	_nAgentByBus.transferCPU();
	VoltageLimitReal.transferCPU();
	Pmin.transferCPU();
	Pmax.transferCPU();
	Pbmax.transferCPU();
	Pbmin.transferCPU();
	Pb.transferCPU();
	Cost1.transferCPU();
	Cost2.transferCPU();

	if (_iterGlobal == 0) {
		std::cout << "algorithm not launch" << std::endl;
	}
	else if (_iterGlobal < _iterG) {
		std::cout << "method " << _name << " converged in " << _iterGlobal << " iterations." << std::endl;
		std::cout << "Converged in " << (float) timeOPF / CLOCKS_PER_SEC << " seconds" << std::endl;

	}
	else {
		std::cout << "method " << _name << " not converged in " << _iterGlobal << " iterations." << std::endl;
		std::cout << "time taken " << (float) timeOPF / CLOCKS_PER_SEC << " seconds" << std::endl;
	}
	std::cout << "The power error of this state is (constraint) " << resF.get(0, _iterGlobal / _stepG) << " and convergence " << resF.get(1, _iterGlobal / _stepG) << std::endl;
	std::cout << "===============================================================|" << std::endl;
	std::cout << "      System Summary                                           |" << std::endl;
	std::cout << "===============================================================|" << std::endl;
	std::cout << "Buses            " << _nBus << std::endl;
	std::cout << "Branches         " << _nLine << std::endl;
	std::cout << "Agent            " << _nAgent << std::endl;
	std::cout << "Ploss            " << getPLoss() << std::endl;
	std::cout << "Qloss            " << getQLoss() << std::endl;


	std::cout << std::endl << std::endl;
	
	std::cout << "========================================================================================================|" << std::endl;
	std::cout << "      Bus Data                                                                                          |" << std::endl;
	std::cout << "========================================================================================================|" << std::endl;
	std::cout << " Bus |    Voltage  |   Power = Generation  + Load    |                Mu voltage and power              |" << std::endl;
	std::cout << "  #  |     Mag(pu) |    P (pu)      |      Q (pu)    |     V (pu)     |      P (pu)    |      Q (pu)    |" << std::endl;
	std::cout << "-----|-------------|----------------|----------------|----------------|----------------|----------------|" << std::endl;

		
	for (int b = 0; b < _nBus; b++) {
		int begining = _indiceBusBegin.get(b, 0);
	std::cout << std::setw(5) << b << "|" << std::setw(12) << sqrt(X.get(begining + 3,0)) << " |" << std::setw(16)
			<< Pb.get(b, 0) << "|" << std::setw(16) << Pb.get(b, 0)
			<< "|" << std::setw(16) << Mu.get(begining + 3, 0) << "|" << std::setw(16)
			<< Mu.get(begining, 0) << "|" << std::setw(16) << Mu.get(begining + 1, 0) << "|" << std::endl;

	}
	std::cout << std::endl << std::endl;
	std::cout << "========================================================================================================|" << std::endl;
	std::cout << "      Line Data                                                                                         |" << std::endl;
	std::cout << "========================================================================================================|" << std::endl;
	std::cout << " Line |    From     |    To      |                           Upstream flow                              |" << std::endl;
	std::cout << "  #   |    Bus      |    Bus     |    P (pu)      |    Q (pu)      |     l (pu)     |     Loss (pu)     |" << std::endl;
	std::cout << "------|-------------|------------|----------------|----------------|----------------|-------------------|" << std::endl;

	for (int l = 0; l < _nLine; l++) {
		int b = l + 1;
		int begining = _indiceBusBegin.get(b, 0);
		std::cout << std::setw(6) << l << "|" << std::setw(12) << CoresLineBus.get(l, 0) << " |" << std::setw(12)
			<< CoresLineBus.get(l, 1) << "|" << std::setw(16) << X.get(begining + 0, 0)
			<< "|" << std::setw(16) << X.get(begining + 1, 0) << "|" << std::setw(16)
			<< X.get(begining + 2, 0) << "|" << std::setw(19) << X.get(begining + 2, 0) * ZsRe.get(l, 0) << "|" << std::endl;
	}
	std::cout << std::endl << std::endl;
	std::cout << "========================================================================================================|" << std::endl;
	std::cout << "     Constraints                                                                                        |" << std::endl;
	std::cout << "========================================================================================================|" << std::endl;
	std::cout << " Bus | Voltage | Voltage | Voltage |        Power Injection          |          Power Injection         |" << std::endl;
	std::cout << "  #  | Mag(pu) | MIN(pu) |  MAX(pu)|  P (pu) | Pmin (pu) | Pmax (pu) |  Q (pu)  | Qmin (pu) | Qmax (pu) |" << std::endl;
	std::cout << "-----|---------|---------|---------|---------|-----------|-----------|----------|-----------|-----------|" << std::endl;
	

	for (int b = 0; b < _nBus; b++) {
		int begining = _indiceBusBegin.get(b, 0);
		std::cout << std::setw(5) << b << "|" << std::setw(8) << sqrt(Y.get(begining + 3, 0)) << " |" << std::setw(9)
			<< VoltageLimitReal.get(0, b) << "|" << std::setw(9) << VoltageLimitReal.get(1, b)
			<< "|" << std::setw(9) << Pb.get(b, 0) << "|" << std::setw(11)
			<< Pbmin.get(b, 0) << "|" << std::setw(11) << Pbmax.get(b, 0)  << "|" << std::setw(10) << Pb.get(b + _nBus, 0)
			<< "|" << std::setw(11) << Pbmin.get(b + _nBus, 0) << "|" << std::setw(11) << Pbmax.get(b + _nBus, 0) << "|" << std::endl;

	}
	std::cout << std::endl << std::endl;
	std::cout << "========================================================================================================|" << std::endl;
	std::cout << "      Agent Data                                                                                        |" << std::endl;
	std::cout << "========================================================================================================|" << std::endl;
	std::cout << " Agent |  Bus  |  Cost   |  Cost   |        Power Injection          |          Power Injection         |" << std::endl;
	std::cout << "  #    |   #   |  a (pu) |  b (pu) |  P (pu) | Pmin (pu) | Pmax (pu) |  Q (pu)  | Qmin (pu) | Qmax (pu) |" << std::endl;
	std::cout << "-------|-------|---------|---------|---------|-----------|-----------|----------|-----------|-----------|" << std::endl;

	for (int n = 0; n < _nAgent; n++) {
		int b = _CoresBusAgent.get(n, 0);
		std::cout << std::setw(7) << n << "|" << std::setw(7) << b << "|" << std::setw(8) << Cost1.get(n,0) << " |" << std::setw(9)
			<< Cost2.get(n, 0) << "|" << std::setw(9) << Pn.get(n,0) << "|" << std::setw(11)
			<< Pmin.get(n, 0) << "|" << std::setw(11) << Pmax.get(n, 0) << "|" << std::setw(10) << Pn.get(n + _nAgent, 0)
			<< "|" << std::setw(11) << Pmin.get(n + _nAgent, 0) << "|" << std::setw(11) << Pmax.get(n + _nAgent, 0) << "|" << std::endl;
	}


	std::cout << "========================================================================================================|" << std::endl;
	std::cout << "                      END PRINT                                                                         |" << std::endl;
	std::cout << "========================================================================================================|" << std::endl;

}





template <unsigned int _blockSizeSmall>
__global__ void updateChatGPU2(float* Chat, float* Y, float* MU, float* nChild, float* Ancestor, float* posChild, float* Childs, float* indiceBusBegin, float* indiceChildBegin, float* CoresChatBegin, float* nAgentByBus, float _rho, int nBus) {

	int bus = blockIdx.x;
	int index = threadIdx.x;
	int step = blockDim.x;
	int begin = CoresChatBegin[bus];

	__shared__ float shArr[_blockSizeSmall]; // c'est grand pour pas grand chose...
	

	int indice = indiceBusBegin[bus];
	int indiceChild = (bus < (nBus - 1)) ? indiceChildBegin[bus] : 0;
	int nb = nChild[bus];
	int Ai = Ancestor[bus];
	int nAgent = nAgentByBus[bus];
	int c = posChild[bus];
	float var = 0;
	int borne = 4 + 2 * nAgent;

	if (index < borne) {
		//float Phat, Qhat, lhat, vihat, pnhat..., qnhat...;
		var =  Y[indice + index] / (1 + (index < 4)) - MU[indice + index] / ((1 + (index < 4)) * _rho);
		if (bus > 0) {
			if (index < 3) {
				int nAi = nChild[Ai];
				int nAgentAi = nAgentByBus[Ai];
				int indiceAncBus = indiceBusBegin[Ai] + 5 + 2 * nAgentAi +  nAi * index + c;
				//var = indiceAncBus;
				var += Y[indiceAncBus] / 2  - MU[indiceAncBus] / (2 * _rho); 
			}			
		}
	}
	float vhat = 0;
	float muhat = 0;
	for (int i = index; i < nb; i += step) {
		int Bus2 = Childs[indiceChild + i];
		int indiceBusChild = indiceBusBegin[Bus2];
		int nAgent2 = nAgentByBus[Bus2];
		muhat += MU[indiceBusChild + 4 + 2 * nAgent2]; // pas du tout coalescent
		vhat += Y[indiceBusChild + 4 + 2 * nAgent2]; // pas du tout coalescent
	}
	shArr[index] = vhat / (nb + 1) - muhat / (_rho * (nb + 1));
	__syncthreads();
	for (int size = _blockSizeSmall / 2; size > 0; size /= 2) { //uniform
		if (index < size) {
			shArr[index] += shArr[index + size];
		}
		__syncthreads();
	}

	if (index < borne) {
		if (index == 3) {
			var = shArr[0] + Y[indice + 3] / (nb + 1) - MU[indice + 3] / (_rho * (nb + 1)); //shArr[0];
		}
		Chat[begin + index] = var; // coalescent  !!!!
	}
}


