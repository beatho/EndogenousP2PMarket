#include "hip/hip_runtime.h"
#include "../head/MatrixGPU.cuh" 

#define MAX(X, Y) X * (X >= Y) + Y * (Y > X)
const int warpSize = 32;

#define CHECK_CUDA_ERROR_MAT(val) checkMat((val), #val, __FILE__, __LINE__);
#define CHECK_LAST_CUDA_ERROR_MAT() checkMatLast(__FILE__, __LINE__);

template <typename T>
void checkMat(T err, char const* const func, char const* const file,
    int const line)
{
    if (err != hipSuccess)
    {
        std::cerr << "CUDA Runtime Error at: " << file << ":" << line
            << std::endl;
        std::cerr << hipGetErrorString(err) << " " << func << std::endl;
        std::exit(EXIT_FAILURE);
    }
}
void checkMatLast(char const* const file, int const line)
{
    hipError_t err{ hipGetLastError() };
    if (err != hipSuccess)
    {
        std::cerr << "CUDA Runtime Error at: " << file << ":" << line
            << std::endl;
        std::cerr << hipGetErrorString(err) << std::endl;
        std::exit(EXIT_FAILURE);
    }
}

float MatrixGPU::rand1()
{
    float a = (float)(rand()) / ((float)(RAND_MAX));
    return a;
}


///////////////////////////////////////////////////////////////////////////////
// Constructor
///////////////////////////////////////////////////////////////////////////////
MatrixGPU::MatrixGPU() {
#ifdef DEBUG_CONSTRUCTOR
    std::cout << "contructeur appele" << std::endl;
#endif
    _row = 0;
    _column = 0;
    _N = _row * _column;
     _numBlocks = MAX(ceil((_N + _blockSize - 1) / _blockSize),1);
}

MatrixGPU::MatrixGPU(int l, int c, float value, bool pos)
{
#ifdef DEBUG_CONSTRUCTOR
    std::cout << "contructeur parametre appele" << std::endl;
    std::cout << _matrixCPU << std::endl;
#endif
    _row = l;
    _column = c;
    _N = _row * _column;
    _numBlocks = MAX(ceil((_N + _blockSize - 1) / _blockSize), 1);
    if (pos) {
        if (_N > 0) {
            hipMalloc((void**)&_matrixGPU, sizeof(float) * _N);
            setGPU << <_numBlocks, _blockSize >> > (_matrixGPU, value, _N);
        }
        _GPU = true;
    }
    else {
        if (_N > 0) {
            _matrixCPU = new float[_N];
        }
        for (int elem = 0; elem < _N; elem++) {
            _matrixCPU[elem] = value;
        }
    }
#ifdef DEBUG_CONSTRUCTOR
    std::cout << _matrixGPU << std::endl;
#endif
}

MatrixGPU::MatrixGPU(const MatrixCPU& m, bool pos)
{
    _row = m.getNLin();
    _column = m.getNCol();
    _N = _row * _column;
    _numBlocks = MAX(ceil((_N + _blockSize - 1) / _blockSize), 1);

    if (pos) {
        _GPU = true;
        if (_N > 0) {
            hipMalloc((void**)&_matrixGPU, sizeof(float) * _row * _column);
            hipMemcpy(_matrixGPU, m._matrixCPU, sizeof(float) * _row * _column, hipMemcpyHostToDevice);
        }
    }
    else {
        if (_N > 0) {
            _matrixCPU = new float[_row * _column];
            memcpy(_matrixCPU, m._matrixCPU, _row * _column * sizeof(float));
        }
    }
    
}

MatrixGPU::MatrixGPU(const MatrixGPU & m)
{
#ifdef DEBUG_CONSTRUCTOR
    std::cout << "contructeur recopie appele" << std::endl;
#endif
    _row = m.getNLin();
    _column = m.getNCol();
    _N = _row * _column;
    _numBlocks = MAX(ceil((_N + _blockSize - 1) / _blockSize), 1);

    if (m.getPos()) {
        if (_N > 0) {
            hipMalloc((void**)&_matrixGPU, sizeof(float) * _row * _column);
            setGPU << <_numBlocks, _blockSize >> > (_matrixGPU, m._matrixGPU, _N);
        }
        _GPU = true;
    }
    else {
        if (_N > 0) {
            _matrixCPU = new float[_row * _column];
            memcpy(_matrixCPU, m._matrixCPU, _row * _column * sizeof(float));
        }
    }
}

MatrixGPU::MatrixGPU(const MatrixGPUD& m)
{
    _row = m.getNLin();
    _column = m.getNCol();
    _N = _row * _column;
    _numBlocks = MAX(ceil((_N + _blockSize - 1) / _blockSize), 1);

    if (m.getPos()) {
        if (_N > 0) {
            hipMalloc((void**)&_matrixGPU, sizeof(float) * _row * _column);
            setGPUFD << <_numBlocks, _blockSize >> > (_matrixGPU, m._matrixGPU, _N);
        }
        _GPU = true;
}
    else {
        if (_N > 0) {
            _matrixCPU = new float[_row * _column];
        }
        for (int i = 0; i < _row; i++) {
            for (int j = 0; j < _column; j++)
            {
                set(i, j, m.get(i, j));
            }
        }
    }
}

MatrixGPU& MatrixGPU::operator=(const MatrixGPU& m)
{
#ifdef DEBUG_CONSTRUCTOR
    std::cout << "contructeur operateur = appele" << std::endl;
#endif
    if (_row == m.getNLin() && _column == m.getNCol()) {
        //matrix already has the good size no free needed
        if (getPos()) {
            if (m.getPos()) {
                setGPU << <_numBlocks, _blockSize >> > (_matrixGPU, m._matrixGPU, _N);
            }
            else {
                if (_N > 0) {
                    hipMemcpy(_matrixGPU, m._matrixCPU, sizeof(float) * _row * _column, hipMemcpyHostToDevice);
                }
            }
        }
        else {
            if (m.getPos()) {
                if (_N > 0) {
                    hipMemcpy(_matrixCPU, m._matrixGPU, sizeof(float) * _row * _column, hipMemcpyDeviceToHost);
                }
            }
            else {
                if (_N > 0) {
                    memcpy(_matrixCPU, m._matrixCPU, _row * _column * sizeof(float));
                }
            }
        }
    }
    else {
        _row = m.getNLin();
        _column = m.getNCol();
        _N = _row * _column;
        _numBlocks = MAX(ceil((_N + _blockSize - 1) / _blockSize), 1);
        _GPU = false;
        if (_matrixGPU) {
            hipFree(_matrixGPU);
            _matrixGPU = nullptr;
        }
        DELETEA(_matrixCPU);
        if (m.getPos()) {
            if (_N > 0) {
                hipMalloc((void**)&_matrixGPU, sizeof(float) * _row * _column);
                setGPU << <_numBlocks, _blockSize >> > (_matrixGPU, m._matrixGPU, _N);
            }
            _GPU = true;
        }
        else {
            if (_N > 0) {
                _matrixCPU = new float[_row * _column];
                memcpy(_matrixCPU, m._matrixCPU, _row * _column * sizeof(float));
            }
        }
    }
   
    return *this;
}

MatrixGPU& MatrixGPU::operator=(const MatrixGPUD& m)
{
    if (_row == m.getNLin() && _column == m.getNCol()) {
        //matrix already has the good size no free needed
        if (getPos()) {
            if (m.getPos()) {
                setGPUFD << <_numBlocks, _blockSize >> > (_matrixGPU, m._matrixGPU, _N);
            }
            else {
                if (_matrixCPU == nullptr) {
                    _matrixCPU = new float[_row * _column];
                }
                for (int i = 0; i < _row; i++) {
                    for (int j = 0; j < _column; j++)
                    {
                        set(i, j, m.get(i, j));
                    }
                }
                transferGPU();
            }
        }
        else {
            if (m.getPos()) {
                if (_matrixGPU == nullptr) {
                    hipMalloc((void**)&_matrixGPU, sizeof(float) * _row * _column);
                }
                setGPUFD << <_numBlocks, _blockSize >> > (_matrixGPU, m._matrixGPU, _N);
                _GPU = true;
                transferCPU();
            }
            else {
                for (int i = 0; i < _row; i++) {
                    for (int j = 0; j < _column; j++)
                    {
                        set(i, j, m.get(i, j));
                    }
                }
            }
        }
    }
    else {
        _row = m.getNLin();
        _column = m.getNCol();
        _N = _row * _column;
        _numBlocks = MAX(ceil((_N + _blockSize - 1) / _blockSize), 1);
        if (_matrixGPU) {
            hipFree(_matrixGPU);
            _matrixGPU = nullptr;
        }
        DELETEA(_matrixCPU);
        if (m.getPos()) {
            hipMalloc((void**)&_matrixGPU, sizeof(float) * _row * _column);
            setGPUFD << <_numBlocks, _blockSize >> > (_matrixGPU, m._matrixGPU, _N);
            _GPU = true;
        }
        else {
            _matrixCPU = new float[_row * _column];
            for (int i = 0; i < _row; i++) {
                for (int j = 0; j < _column; j++)
                {
                    set(i, j, m.get(i, j));
                }
            }
        }
    }
    return *this;
}

MatrixGPU& MatrixGPU::operator=(const MatrixCPU& m)
{
    if (_row == m.getNLin() && _column == m.getNCol()) {
        //matrix already has the good size no free needed
        if (getPos()) {
            if (_N > 0) {
                hipMemcpy(_matrixGPU, m._matrixCPU, sizeof(float) * _row * _column, hipMemcpyHostToDevice);
            }
        }
        else {
            if (_N > 0) {
                memcpy(_matrixCPU, m._matrixCPU, _row * _column * sizeof(float));
            }
        }
    }
    else {
        _row = m.getNLin();
        _column = m.getNCol();
        _N = _row * _column;
        _numBlocks = MAX(ceil((_N + _blockSize - 1) / _blockSize),1);
        if (_matrixGPU) {
            hipFree(_matrixGPU);
            _matrixGPU = nullptr;
        }
        DELETEA(_matrixCPU);

        if (getPos()) {
            if (_N > 0) {
                hipMalloc((void**)&_matrixGPU, sizeof(float) * _row * _column);
                hipMemcpy(_matrixGPU, m._matrixCPU, sizeof(float) * _row * _column, hipMemcpyHostToDevice);
            }
            _GPU = true;
        }
        else
        {
            if (_N > 0) {
                _matrixCPU = new float[_row * _column];
                memcpy(_matrixCPU, m._matrixCPU, _row * _column * sizeof(float));
            }
        } 
    }
    return *this;
}

void MatrixGPU::preallocateReduction()
{
    CHECK_LAST_CUDA_ERROR();
    if (preallocation) {
        hipHostFree(_preallocationFloat);
        hipFree(_preallocation);
        preallocation = false;
    }
    hipError_t c;
    int counter = 0;
     /* do
    {
        c = hipHostAlloc(&_preallocationFloat, sizeof(float), hipHostMallocDefault);
        counter++;
    } while (_preallocationFloat == nullptr && counter < 10);
   
     
    if (_preallocationFloat == nullptr) {
            
        std::cout << "prealocation echouer ? " << c << std::endl;
        std::cout <<  hipGetErrorName(c) << std::endl;
    }/
     if (c == 700) {
        std::cout << "c=700" << std::endl;
        exit(-1);
     }*/
     do
     {
         c = hipMalloc((void**)&_preallocation, sizeof(float) * _numBlocks);
         counter++;
         hipDeviceSynchronize();
     } while (_preallocation == nullptr && counter < 20);

    if (_preallocation == nullptr) {
        std::cout << _row << " " << _column << " " << _blockSize << std::endl;
        std::cout << "prealocation echouer ? " << c << " " << _numBlocks  <<std::endl;
        std::cout << hipGetErrorName(c) << std::endl;
    }
    if (c == 700) {
        exit(-1);
    }/**/
    
    preallocation = true;
    setGPU <<<_numBlocks, _blockSize >>> (_preallocation, 0.0f, _numBlocks);
}

void MatrixGPU::transferGPU()
{
    if (!_GPU) {
        if (!_matrixGPU) {
            hipMalloc((void**)&_matrixGPU, sizeof(float) * _row * _column);
        }
        hipMemcpy(_matrixGPU, _matrixCPU, sizeof(float) * _row * _column, hipMemcpyHostToDevice);
        //DELETEA(_matrixCPU);
        _GPU = true;
    }
    else {
        throw std::domain_error("already in the GPU");
    }
    
}

void MatrixGPU::transferCPU()
{
    
    if (_GPU) {
        
        if (!_matrixCPU) {
            
            _matrixCPU = new float[_row * _column];
        }
        hipMemcpy(_matrixCPU, _matrixGPU, sizeof(float) * _row * _column, hipMemcpyDeviceToHost);
        //hipFree(_matrixGPU);
        //_matrixGPU = nullptr;
        _GPU = false;
    }
    else {
        std::cout << "transferCPU : already in the CPU " << _GPU <<std::endl;
        throw std::domain_error("already in the CPU");
    }

}

///////////////////////////////////////////////////////////////////////////////
// Getter
///////////////////////////////////////////////////////////////////////////////
 float MatrixGPU::get(int i, int j, bool verbose) const
{
    //std::cout << "hey de taille " << _row << " " << _column << "pos "<< i <<" "<< j << std::endl;
    if ((i >= _row) || ( j >= _column) || (i < 0) || ( j < 0)) {
        std::cout << "get" << _row << " " << _column << " " << i << " " << j << std::endl;
        throw std::out_of_range("index out of bounds");
    }
    if (_GPU) {
        float value;
        hipMemcpy(&value, _matrixGPU + i*_column+j, sizeof(float), hipMemcpyDeviceToHost);
        if (verbose) {
            std::cout << " Warning matrix on GPU" << std::endl;
        }
        return value;
        //throw std::invalid_argument("Matrix on GPU");
    }
    else {
        return _matrixCPU[i * _column + j];
    }
}

int MatrixGPU::getNCol() const
{
    return _column;
}

int MatrixGPU::getNLin() const
{
    return _row;
}

void MatrixGPU::getCol(MatrixGPU* col, int numCol, int offset)
{
    if (numCol < 0 || numCol >= _column) {
        throw std::out_of_range("index out of bounds");
    }
    if (offset < 0 || offset >= _row) {
        throw std::out_of_range("index out of bounds");
    }
    if (col->getNLin() != _row) {
        throw std::invalid_argument("not the same dimension");
    }
    if (col->getNCol() != 1) {
        throw std::invalid_argument("must be a column vector");
    }

    if (!_GPU && !col->getPos()) {
        for (int i = 0; i < offset; i++) {
            col->set(i, 0, 0);
        }
        for (int i = offset; i < _row; i++) {
            col->set(i, 0, get(i, numCol));
        }
    }
    else if (_GPU && col->getPos()) {
        setColGPU <<< _numBlocks, _blockSize >>> (col->_matrixGPU, _matrixGPU, numCol, _column, _row, offset);
    } else {
        throw std::invalid_argument("getCol Matrix not at the same place");
    }

}


bool  MatrixGPU::getPos() const
{
    return _GPU;
}
bool MatrixGPU::dim(MatrixGPU* m) const
{ 
    return ((_row == m->getNLin()) && (_column == m->getNCol()));
}


bool MatrixGPU::isEqual(MatrixGPU* m, float pre) const
{
    if (!dim(m)) {
        throw std::invalid_argument("is Equal : not the same dimension");
    }
    else {
        if (_GPU || m->getPos()) {
            throw std::invalid_argument("is Equal : Matrix on GPU");
        }
        else {
            for (int i = 0; i < _row; i++) {
                for (int j = 0; j < _column; j++) {
                    if (fabs(get(i, j) - m->get(i, j)) > pre) {
                        return false;
                    }
                }
            }
        }
    }
    return true;
}

void MatrixGPU::toMatCPU(MatrixCPU& m) const // passer m en paramètre
{
    if (m.getNCol() != _column || m.getNLin() != _row) {
        m.setSize(_row, _column);
    }
    if (_GPU) {
        hipMemcpy(m._matrixCPU, _matrixGPU, sizeof(float) * _row * _column, hipMemcpyDeviceToHost);
    }
    else {
        memcpy(m._matrixCPU, _matrixCPU, sizeof(float) * _row * _column );
        /*for (int i = 0; i < _row; i++) {
            for (int j = 0; j < _column; j++) 
            {
                m.set(i, j, get(i, j));
            }
        }*/
    }
}

void MatrixGPU::toMatGPUD(MatrixGPUD& m) const
{
    if (m.getNCol() != _column || m.getNLin() != _row) {
        std::cout << "pas de bonne taille" << std::endl;
        m.setSize(_row, _column);
    }
    if (_GPU) {
        if (!m.getPos()) {
            std::cout << "transfert GPU" << std::endl;
            m.transferGPU();
        }
       
        setGPUDF << <_numBlocks, _blockSize >> > (m._matrixCPU, _matrixGPU, _N);
        m.transferCPU();
        m.display();
        m.transferGPU();

    }
    else {
        if (m.getPos()) {
            m.transferCPU();
        }
        for (int i = 0; i < _row; i++) {
            for (int j = 0; j < _column; j++)
            {
                m.set(i, j, get(i, j));
            }
        }
    }

}



///////////////////////////////////////////////////////////////////////////////
// Setter
///////////////////////////////////////////////////////////////////////////////
 void MatrixGPU::set(int i, int j, float value, bool force)
{
    if ((i >= _row) || (j >= _column) || (i < 0) || (j < 0)) {
        std::cout << _row << " " << _column << " " << i << " " << j << std::endl;
        throw std::out_of_range("set : index out of bounds");
    }
    if (_GPU && !force) {
        throw std::invalid_argument("set : Matrix on GPU");
    }
    else if (_GPU && force) {
        setGPUunique <<< 1, 1 >>> (_matrixGPU, value, i * _column + j);
    }
    else {
        //std::cout << "changement de valeur " << value << " en " << i << " " << j << std::endl;
        _matrixCPU[i * _column + j] = value;
    }
}

 void MatrixGPU::setEyes(float value)
 {
     if (!_GPU) {
         int N = _row * (_row < _column) + _column * (_column <= _row);

         for (int i = 0; i < _row; i++) {
             for (int j = 0; j < _column; j++)
             {
                 if (i == j) {
                     set(i, j, value);
                 }
                 else {
                     set(i, j, 0);
                 }  
             }
         }
     }
     else {
         setEyesGPU<<< _numBlocks, _blockSize >>>(_matrixGPU, value, _column, _row);
     }
     
 }

 void MatrixGPU::setEyes(MatrixGPU* m)
 {
     if (m->getNLin() != _row || _row != _column || m->getNCol() != 1) 
     {
         throw std::invalid_argument("not the good dimension");
     }
     if (_GPU && m->getPos()) {
         setEyesGPU <<< _numBlocks, _blockSize >> > (_matrixGPU, m->_matrixGPU, _column, _row);

     }
     else if (!_GPU && !(m->getPos()))
     {
         for (int i = 0; i < _row; ++i)
         {
            this->set(i, i, m->get(i, 0));
         }
     }
     else {
         throw std::invalid_argument("setEyes Matrix not at the same place");
     }



 }

void MatrixGPU::set(MatrixGPU* m, bool synchrone, hipStream_t stream)
{
   
    if (!dim(m)) {
        throw std::invalid_argument("not the same dimension");
    }
    if (_GPU && m->getPos()) {
        if (synchrone) {
            setGPU <<<_numBlocks, _blockSize >> > (_matrixGPU, m->_matrixGPU, _N);
        }
        else {
            setGPU <<< _numBlocks, _blockSize, 0, stream>>> (_matrixGPU, m->_matrixGPU, _N);
        }
        
    }
    else if (!_GPU && !(m->getPos())) 
    {
        for (int i = 0; i < _row;++i)
        {
            for (int j = 0;j < _column;++j)
            { 
                this->set(i, j, m->get(i, j));
            }
        }
    }
    else {
        throw std::invalid_argument("set Matrix not at the same place");
    }

}

void MatrixGPU::set(MatrixCPU* m)
{
    if (m->getNCol() != _column || m->getNLin() != _row) {
        throw std::invalid_argument("not the same dimension");
    }
    
    if (getPos()) {
        hipMemcpy(_matrixGPU, m->_matrixCPU, sizeof(float) * _row * _column, hipMemcpyHostToDevice);
    }
    else {
        memcpy(_matrixCPU, m->_matrixCPU, _row * _column * sizeof(float));
    }
}

void MatrixGPU::set(double value)
{
   
    if (_GPU) {
        setGPU << <_numBlocks, _blockSize >> > (_matrixGPU, value, _N);
    
    }
    else {
        for (int elem = 0; elem < _N; elem++) {
            _matrixCPU[elem] = value;
        }
    }
}

void MatrixGPU::setTrans(MatrixGPU* m)
{
    if (_column != m->getNLin() || _row != m->getNCol()) {
        std::cout << _row << " " << _column << " " << m->getNLin() <<" " << m->getNCol() <<std::endl;
        throw std::invalid_argument(" setTans : not the same transposed dimension");
    }
    if (getPos() && m->getPos()) {
        if (_N > 0) {
            setTransGPU << <_numBlocks, _blockSize >> > (_matrixGPU, m->_matrixGPU, _column, _row);
        }
    }
    else if (!getPos() && !(m->getPos()))
    {
        for (int i = 0; i < _row; i++) {
            for (int j = 0; j < _column; j++) {
                set(i, j, m->get(j, i));
            }
        }
    }
    else {
        throw std::invalid_argument("setTrans Matrix not at the same place");
    }

}


void MatrixGPU::setRand(float eps)
{
    //exit(1);
    if (_GPU) {

        /*hiprandGenerator_t gen;
        hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
        hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL);
        hiprandGenerateUniform(gen, _matrixGPU, _N);
        hiprandDestroyGenerator(gen);*/
        
        hiprandState* state = nullptr;
        hipMalloc((void**)&state, _N * sizeof(hiprandState));
        setup_kernel <<<_numBlocks, _blockSize >>> (state);
        generate_kernel << <_numBlocks, _blockSize >> > (state, _matrixGPU, eps, _N);
        //throw std::invalid_argument("Matrix on GPU");
    }
    else {
        int N = _column * _row;
        for (int elem = 0; elem < N; elem++) {
            _matrixCPU[elem] = 2 * (rand1() - 0.5) * eps;

        }
    }
    
}

void MatrixGPU::setBloc(int iBegin, int iEnd, int jBegin, int jEnd, MatrixGPU* m)
{
    if ((iBegin < 0) || (jBegin < 0) || iEnd > _row || jEnd > _column) {
        std::cout << iBegin << " " << iEnd  << " " << jBegin << " " << jEnd << " " << _row << " " << _column << std::endl;
        throw std::out_of_range(" setBloc : index out of bounds"); 
    } if ((iBegin > iEnd) || (jBegin > jEnd)) {
        throw std::invalid_argument("setBloc : xBegin must be smaller than xEnd");
    } if (m->getNLin() != (iEnd - iBegin) || m->getNCol() != (jEnd - jBegin)) {
        throw std::invalid_argument("setBloc : not the same dimension");
    }
    if (!_GPU && !(m->getPos())) {
        int row = 0;

        for (int i = iBegin; i < iEnd; i++) {
            int col = 0;
            for (int j = jBegin; j < jEnd;j++) {
                set(i, j, m->get(row, col));
                col = col + 1;
            }
            row = row + 1;
        }
    }
    else if (getPos() && (m->getPos())) {
        /*const int nThread = 16;
         const int bx = (jEnd - jBegin + nThread - 1) / nThread;
         const int by = (iEnd - iBegin + nThread - 1) / nThread;
         dim3 gridBlock(bx, by);
         dim3 dimBlock(nThread, nThread);*/
        SetBlocGPU << <m->_numBlocks, m->_blockSize >> > (_matrixGPU, m->_matrixGPU, iBegin, iEnd, jBegin, jEnd, _column);
    }
    else {
       
        throw std::invalid_argument("setBloc : Matrix not at the same place");
    }
}
void MatrixGPU::setBloc(int iBegin, int iEnd, int jBegin, int jEnd, MatrixGPU* m, float factor)
{
    if ((iBegin < 0) || (jBegin < 0) || iEnd > _row || jEnd > _column) {
        throw std::out_of_range("index out of bounds");
    } if ((iBegin > iEnd) || (jBegin > jEnd)) {
        throw std::invalid_argument("xBegin must be smaller than xEnd");
    } if (m->getNLin() != (iEnd - iBegin) || m->getNCol() != (jEnd - jBegin)) {
        throw std::invalid_argument("not the same dimension");
    }
    if (!_GPU && !(m->getPos())) {
        int row = 0;

        for (int i = iBegin; i < iEnd; i++) {
            int col = 0;
            for (int j = jBegin; j < jEnd; j++) {
                set(i, j, factor * m->get(row, col));
                col = col + 1;
            }
            row = row + 1;
        }
    }
    else if (getPos() && (m->getPos())) {
        /*const int nThread = 16;
        const int bx = (jEnd - jBegin + nThread - 1) / nThread;
        const int by = (iEnd - iBegin + nThread - 1) / nThread;
        dim3 gridBlock(bx, by);
        dim3 dimBlock(nThread, nThread);*/
        SetBlocGPU <<<m->_numBlocks, m->_blockSize >> > (_matrixGPU, m->_matrixGPU, iBegin, iEnd, jBegin, jEnd, _column, factor);
    }
    else {
        throw std::invalid_argument("setBloc Matrix not at the same place");
    }
}
void MatrixGPU::setBloc(int iBegin, int iEnd, int jBegin, int jEnd, MatrixCPU* m)
{
    if ((iBegin < 0) || (jBegin < 0) || iEnd > _row || jEnd > _column) {
        std::cout << _row << " " << _column << " " << iEnd << " " << jEnd << std::endl;
        throw std::out_of_range("setBloc: index out of bounds");
    } if ((iBegin > iEnd) || (jBegin > jEnd)) {
        throw std::invalid_argument("setBloc : xBegin must be smaller than xEnd");
    } if (m->getNLin() != (iEnd - iBegin) || m->getNCol() != (jEnd - jBegin)) {
        throw std::invalid_argument("setBloc : not the same dimension");
    }
    
    if (!_GPU) {
        int row = 0;
        
        for (int i = iBegin; i < iEnd; i++) {
            int col = 0;
            for (int j = jBegin; j < jEnd;j++) {
               // if (iBegin == 44597 && row > 911 && col > 700) {  
                set(i, j, m->get(row, col));
                col = col + 1;
            }
          
            row = row + 1;
        }
    }
    else {
        throw std::domain_error("setBloc : Matrix on GPU");
    }
}


void MatrixGPU::setBloc(int iBegin, int iEnd, int jBegin, int jEnd, float value)
{
    if ((iBegin < 0) || (jBegin < 0) || iEnd > _row || jEnd > _column) {
        std::cout << iBegin << " " << iEnd << " " << jBegin << " " << jEnd << " " << _row << " " << _column << std::endl;
        throw std::out_of_range(" setBloc : index out of bounds");
    } if ((iBegin >= iEnd) || (jBegin >= jEnd)) {
        throw std::invalid_argument("setBloc : xBegin must be smaller than xEnd");
    } 
    if (!_GPU) {
        int row = 0;

        for (int i = iBegin; i < iEnd; i++) {
            for (int j = jBegin; j < jEnd; j++) {
                set(i, j, value);
            }
        }
    }
    else if (getPos()) {
       
        SetBlocGPU << <_numBlocks, _blockSize >> > (_matrixGPU, value, iBegin, iEnd, jBegin, jEnd, _column);
    }
    else {

        throw std::invalid_argument("setBloc :Matrix not at the same place");
    }
}



void MatrixGPU::swap(MatrixGPU* m)
{
    if (!dim(m)) {
        throw std::invalid_argument("swap : not the same dimension");
    }
    if (_GPU && m->getPos()) {
        float* temp = _matrixGPU;
        _matrixGPU = m->_matrixGPU;
        m->_matrixGPU = temp;

    }
    else if (!_GPU && !(m->getPos())) {
        float* temp = _matrixCPU;
        _matrixCPU = m->_matrixCPU;
        m->_matrixCPU = temp;
    }
    else {
        throw std::invalid_argument("swap : Matrix not at the same place");
    } 
}

void MatrixGPU::replace(float previous, float newValue)
{
    if (_GPU) {
        replaceGPU <<<_numBlocks, _blockSize >> > (_matrixGPU, previous, newValue, _N);
    }
    else {
        for (int i = 0;i < _row;++i)
        {
            for (int j = 0;j < _column;++j)
            {
                if (get(i, j) == previous) {
                    this->set(i, j, newValue);
                }
            }
        }
    }
}

///////////////////////////////////////////////////////////////////////////////
// Addition
///////////////////////////////////////////////////////////////////////////////
void MatrixGPU::add(MatrixGPU* m)
{
    if (!dim(m)) {
        throw std::invalid_argument("add : not the same dimension");
    }
    if (_GPU && m->getPos()) 
    {
        addGPU<<<_numBlocks, _blockSize>>>(_matrixGPU,m->_matrixGPU,_N);
    }
    else if (!_GPU && !(m->getPos()))
    {
        for (int i = 0;i < _row;++i)
        {
            for (int j = 0;j < _column;++j)
            {
                float r = get(i, j) + m->get(i, j);
                this->set(i, j, r);
            }
        }
    } else {
        throw std::invalid_argument("add : Matrix not at the same place");
    } 
}

void MatrixGPU::addVector(MatrixGPU* v)
{
    if (((v->getNCol() != 1) || (v->getNLin() != _row)) && ((v->getNLin() != 1) || (v->getNCol() != _column))) {
        throw std::invalid_argument("addVector : wrong dimension of the vector");
    }
    if (v->getNCol() == 1) {
        if (_GPU && v->getPos()) 
        {
            addVectorGPU1<<<_numBlocks, _blockSize >>>(_matrixGPU, v->_matrixGPU, _column, _N);
        }
        else if ((!_GPU) && !(v->getPos())) {
            for (int i = 0;i < _row;++i)
            {
                for (int j = 0;j < _column;++j)
                {
                    float r = get(i, j) + v->get(i, 0);
                    this->set(i, j, r);
                }
            }
        }
        else {
            throw std::invalid_argument("addVector : Matrix not at the same place");
        } 
    }
    else {
        if (_GPU && v->getPos())
        {
            addVectorGPU2<<<_numBlocks, _blockSize >>>(_matrixGPU, v->_matrixGPU, _column, _N);
        }
        else if ((!_GPU) && !(v->getPos())) {
            for (int i = 0;i < _row;++i)
            {
                for (int j = 0;j < _column;++j)
                {
                    float r = get(i, j) + v->get(0, j);
                    this->set(i, j, r);
                }
            }
        }
        else {
            throw std::invalid_argument("addVector : Matrix not at the same place");
        }
    }
}
void MatrixGPU::add(float c)
{
    if (_GPU) {
        addGPU<<<_numBlocks,_blockSize >>>(_matrixGPU,c, _N);
    }
    else {
        for (int i = 0;i < _row;++i)
        {
            for (int j = 0;j < _column;++j)
            {
                float r = get(i, j) + c;
                this->set(i, j, r);
            }
        }
    }
}

void MatrixGPU::add(MatrixGPU* m1, MatrixGPU* m2)
{
    if (!m1->dim(m2)) {
        throw std::invalid_argument("not the same dimension, fct add, m1 with m2");
    }
    if (!dim(m1)) {
        std::cout << _row << " " << m1->_row << " " << _column << " " << m1->_column << std::endl;
        throw std::invalid_argument("not the same dimension, fct add, this with m1");
    }
    if (_GPU && m1->getPos() && m2->getPos()) 
    {
        addGPU<<<_numBlocks, _blockSize >> > (_matrixGPU, m1->_matrixGPU,m2->_matrixGPU, _N);
    }
    else if (!_GPU && !(m1->getPos()) && !(m2->getPos()))
    {
        for (int i = 0;i < _row;++i)
        {
            for (int j = 0;j < _column;++j)
            {
                float r = m1->get(i, j) + m2->get(i, j);
                this->set(i, j, r);
            }
        }
    }
    else {
        throw std::invalid_argument("add Matrix not at the same place");
    }
    
}
void MatrixGPU::add(MatrixGPU* m, float c)
{
    if (_GPU && m->getPos()) 
    {
        addGPU<<<_numBlocks, _blockSize >>> (_matrixGPU,m->_matrixGPU, c, _N);
    }
    else if ((!_GPU) && !(m->getPos())) 
    {
        for (int i = 0;i < _row;++i)
        {
            for (int j = 0;j < _column;++j)
            {
                float r = m->get(i, j) + c;
                this->set(i, j, r);
            }
        }
    }
    else {
        throw std::invalid_argument("add Matrix not at the same place");
    }
    

}
void MatrixGPU::addTrans(MatrixGPU* m)
{
    MatrixGPU temp(*this);
    if (_row != m->getNCol() && _column != m->getNLin())
    {
        throw std::invalid_argument("addTrans not the same dimension (transpose)");
    }
    if (_GPU && m->getPos())
    {
        addTransGPU<<<_numBlocks, _blockSize >>>(temp._matrixGPU, _matrixGPU, m->_matrixGPU,_column,_row,_N);
    }
    else if (!_GPU && !(m->getPos()))
    {
        for (int i = 0;i < _row;++i)
        {
            for (int j = 0;j < _column;++j)
            {
                float r = get(i, j) + m->get(j, i);
                temp.set(i, j, r);
            }
        }
    }
    else {
        throw std::invalid_argument("addTrans Matrix not at the same place");
    }
    this->set(&temp);
    
}
///////////////////////////////////////////////////////////////////////////////
// subtraction
///////////////////////////////////////////////////////////////////////////////
void MatrixGPU::subtract(MatrixGPU* m1, MatrixGPU* m2)
{
    if (!m1->dim(m2)) {
        throw std::invalid_argument("subtract not the same dimension m1 with m2");
        
    }
    if (!dim(m1)) {
        throw std::invalid_argument("subtract not the same dimension m1 with this");
    }
    if (_GPU && m1->getPos() && m2->getPos())
    {
        substractGPU<<<_numBlocks, _blockSize >> > (_matrixGPU, m1->_matrixGPU, m2->_matrixGPU, _N);
    }
    else if (!_GPU && !(m1->getPos()) && !(m2->getPos()))
    {
        for (int i = 0;i < _row;++i)
        {
            for (int j = 0;j < _column;++j)
            {
                float r = m1->get(i, j) - m2->get(i, j);
                this->set(i, j, r);
            }
        }
    }
    else {
        throw std::invalid_argument(" subtract Matrix not at the same place");
    }
    
}
void MatrixGPU::subtract(MatrixGPU* m)
{
    
    if (!dim(m)) {
        throw std::invalid_argument("subtract not the same dimension");
    }
    if (_GPU && m->getPos())
    {
        substractGPU <<<_numBlocks, _blockSize >> > ( _matrixGPU, m->_matrixGPU, _N);
    }
    else if (!_GPU && !(m->getPos()))
    {
        for (int i = 0;i < _row;++i)
        {
            for (int j = 0;j < _column;++j)
            {
                float r = get(i, j) - m->get(i, j);
                set(i, j, r);
            }
        }
    }
    else {
        throw std::invalid_argument("subtract Matrix not at the same place");
    }
   
    
}
void MatrixGPU::subtractVector(MatrixGPU* v)
{
    if (((v->getNCol() != 1) || (v->getNLin() != _row)) && ((v->getNLin() != 1) || (v->getNCol() != _column))) {
        throw std::invalid_argument( " subtractVector wrong dimension of the vector");
    }
    if (v->getNCol() == 1) {
        if (_GPU && v->getPos())
        {
            substractVectorGPU1 <<<_numBlocks, _blockSize >>>(_matrixGPU, v->_matrixGPU, _column, _N);
        }
        else if ((!_GPU) && !(v->getPos())) {
            for (int i = 0;i < _row;++i)
            {
                for (int j = 0;j < _column;++j)
                {
                    float r = get(i, j) - v->get(i, 0);
                    this->set(i, j, r);
                }
            }
        }
        else {
            throw std::invalid_argument("subtractVector Matrix not at the same place");
        }
    }
    else {
        if (_GPU && v->getPos())
        {
            substractVectorGPU2 <<<_numBlocks, _blockSize >>>(_matrixGPU, v->_matrixGPU, _column, _N);
        }
        else if ((!_GPU) && !(v->getPos())) {
            for (int i = 0;i < _row;++i)
            {
                for (int j = 0;j < _column;++j)
                {
                    float r = get(i, j) - v->get(0, j);
                    this->set(i, j, r);
                }
            }
        }
        else {
            throw std::invalid_argument("subtractVector Matrix not at the same place");
        }
    }

}
void MatrixGPU::subtractTrans(MatrixGPU* m)
{
    if (_row != m->getNCol() && _column != m->getNLin())
    {
        throw std::invalid_argument("subtractTrans not the same dimension (transpose)");
    }
    MatrixGPU temp(*this);
    if (_GPU && m->getPos())
    {
        substractTransGPU <<<_numBlocks, _blockSize >>>(temp._matrixGPU, _matrixGPU, m->_matrixGPU, _column, _row, _N);
    }
    else if (!_GPU && !(m->getPos()))
    {
        for (int i = 0;i < _row;++i)
        {
            for (int j = 0;j < _column;++j)
            {
                float r = get(i, j) - m->get(j, i);
                temp.set(i, j, r);
            }
        }
    }
    else {
        throw std::invalid_argument("subtractTrans Matrix not at the same place");
    }
    this->set(&temp);
}

///////////////////////////////////////////////////////////////////////////////
// Multiplication
///////////////////////////////////////////////////////////////////////////////


void MatrixGPU::multiply(float c)
{
    if (_GPU) {
        multiplyGPU<<<_numBlocks, _blockSize >>> (_matrixGPU, c, _N);
    }
    else {
        for (int i = 0;i < _row;++i)
        {
            for (int j = 0;j < _column;++j)
            {
                float r = get(i, j) * c;
                this->set(i, j, r);
            }
        }
    }
        
}

void MatrixGPU::multiplyMat(MatrixGPU* A, MatrixGPU* B)
{
    if (A->getNLin() != getNLin()) {
        throw std::invalid_argument("multiplyMat result must be compatible with A (row)");
    }
    else if (A->getNCol() != B->getNLin()) {
        throw std::invalid_argument("multiplyMat A must be compatible with B (column with row)");
    }
    else if (getNCol() != B->getNCol()) {
        throw std::invalid_argument("multiplyMat result must be compatible with B (column)");

    }
    if (_GPU && A->getPos() && B->getPos()) { // solution temporaire
        transferCPU();
        A->transferCPU();
        B->transferCPU();
        float r = 0;
        int p = A->getNCol();
        for (int i = 0; i < _row; ++i)
        {
            for (int j = 0; j < _column; ++j)
            {
                r = 0;
                for (int k = 0; k < p; ++k)
                {
                    r += A->get(i, k) * B->get(k, j);
                }
                this->set(i, j, r);
            }
        }
        transferGPU();
        A->transferGPU();
        B->transferGPU();
    }
    else if (!_GPU && !(A->getPos()) && !B->getPos()) {
        float r = 0;
        int p = A->getNCol();
        for (int i = 0; i < _row; ++i)
        {
            for (int j = 0; j < _column; ++j)
            {
                r = 0;
                for (int k = 0; k < p; ++k)
                {
                    r += A->get(i, k) * B->get(k, j);
                }
                this->set(i, j, r);
            }
        }
    }
    else {
        throw std::invalid_argument("multiplyMat Matrix not at the same place");
    }
}

void MatrixGPU::multiply(MatrixGPU* Mat, MatrixGPU* vect, bool trans)
{
    // result = Mat*vect  nLine*nCol = nLine*Taille *Taille * Ncol 
    if (trans) {
        if (getNLin() != 1) {
            throw std::invalid_argument("multiply result must be a row vector ");
        }
        else if (getNCol() != Mat->getNLin()) {
            throw std::invalid_argument("multiply result must be compatible with Mat");
        }
        else if (vect->getNLin() != 1) {
            throw std::invalid_argument("multiply vect must be a row vector ");
        }
        else if (vect->getNCol() != Mat->getNCol()) {
            throw std::invalid_argument("multiply vect must be compatible with Mat");
        }
    }
    else {
        if (getNCol() != 1) {
            throw std::invalid_argument("multiply result must be a column vector ");
        }
        else if (getNLin() != Mat->getNLin()) {
            throw std::invalid_argument("multiply result must have the same row number as the Mat");
        }
        else if (vect->getNCol() != 1) {
            throw std::invalid_argument("multiply vect must be a column vector ");
        }
        else if (vect->getNLin() != Mat->getNCol()) {
            throw std::invalid_argument("multiply vect must be compatible with Mat");
        }
    }
   
    if (_GPU && Mat->getPos() && vect->getPos())
    {
        int numBlock = Mat->getNLin();
        switch (_blockSize) {
        case 512:
            multiplyGPU<512> << <numBlock, _blockSize >> > (_matrixGPU, Mat->_matrixGPU, vect->_matrixGPU, Mat->getNCol());
            break;
        case 256:
            multiplyGPU<256> << <numBlock, _blockSize >> > (_matrixGPU, Mat->_matrixGPU, vect->_matrixGPU, Mat->getNCol());
            break;
        case 128:
            multiplyGPU<128> << <numBlock, _blockSize >> > (_matrixGPU, Mat->_matrixGPU, vect->_matrixGPU, Mat->getNCol());
            break;
        case 64:
            multiplyGPU< 64> << <numBlock, _blockSize >> > (_matrixGPU, Mat->_matrixGPU, vect->_matrixGPU, Mat->getNCol());
            break;
        case 32:
            multiplyGPU< 32> << <numBlock, _blockSize >> > (_matrixGPU, Mat->_matrixGPU, vect->_matrixGPU, Mat->getNCol());
            break;
        case 16:
            multiplyGPU< 16> << <numBlock, _blockSize >> > (_matrixGPU, Mat->_matrixGPU, vect->_matrixGPU, Mat->getNCol());
            break;
        case  8:
            multiplyGPU<  8> << <numBlock, _blockSize >> > (_matrixGPU, Mat->_matrixGPU, vect->_matrixGPU, Mat->getNCol());
            break;
        case  4:
            multiplyGPU<  4> << <numBlock, _blockSize >> > (_matrixGPU, Mat->_matrixGPU, vect->_matrixGPU, Mat->getNCol());
            break;
        case  2:
            multiplyGPU<  2> << <numBlock, _blockSize >> > (_matrixGPU, Mat->_matrixGPU, vect->_matrixGPU, Mat->getNCol());
            break;
        case  1:
            multiplyGPU<  1> << <numBlock, _blockSize >> > (_matrixGPU, Mat->_matrixGPU, vect->_matrixGPU, Mat->getNCol());
            break;
        }
        
    }
    else if (!_GPU && !(Mat->getPos()) && !vect->getPos())
    {
        if (trans) {
            for (int i = 0; i < Mat->getNLin(); ++i)
            {
                float sum = 0;
                for (int j = 0; j < Mat->getNCol(); ++j)
                {
                    sum += Mat->get(i, j) * vect->get(0, j);
                }
                set(0, i, sum);
            }
        }
        else {
            for (int i = 0; i < _row; ++i)
            {
                float sum = 0;
                for (int j = 0; j < Mat->getNCol(); ++j)
                {
                    sum += Mat->get(i, j) * vect->get(j, 0);
                }
                set(i, 0, sum);
            }
        }
    }
    else {
        throw std::invalid_argument("multiply Matrix not at the same place");
    }
}

void MatrixGPU::MultiplyMatTransVec(MatrixGPU* MatToTrans, MatrixGPU* vect, bool rowVector)
{
    // methode tres peu efficace, acces memoire pas coalescent, mais on n'y peut rien...
    if (rowVector) {
        if (getNLin() != 1) {
            throw std::invalid_argument("MultiplyMatTransVec result must be a row vector ");
        }
        else if (getNCol() != MatToTrans->getNCol()) {
            throw std::invalid_argument("MultiplyMatTransVec result must be compatible with Mat");
        }
        else if (vect->getNLin() != 1) {
            throw std::invalid_argument("MultiplyMatTransVec vect must be a row vector ");
        }
        else if (vect->getNCol() != MatToTrans->getNLin()) {
            throw std::invalid_argument("MultiplyMatTransVec vect must be compatible with Mat");
        }
    }
    else {
        if (getNCol() != 1) {
            throw std::invalid_argument("MultiplyMatTransVec result must be a column vector ");
        }
        else if (getNLin() != MatToTrans->getNCol()) {
            throw std::invalid_argument("MultiplyMatTransVec result must have the same row number as the Mat");
        }
        else if (vect->getNCol() != 1) {
            throw std::invalid_argument("MultiplyMatTransVec vect must be a column vector ");
        }
        else if (vect->getNLin() != MatToTrans->getNLin()) {
            throw std::invalid_argument("MultiplyMatTransVec vect must be compatible with Mat");
        }
    }

    if (_GPU && MatToTrans->getPos() && vect->getPos())
    {
        int numBlock = MatToTrans->getNCol();
        switch (_blockSize) {
        case 512:
            multiplyGPUMatVectTrans<512> << <numBlock, _blockSize >> > (_matrixGPU, MatToTrans->_matrixGPU, vect->_matrixGPU, MatToTrans->getNCol());
            break;
        case 256:
            multiplyGPUMatVectTrans<256> << <numBlock, _blockSize >> > (_matrixGPU, MatToTrans->_matrixGPU, vect->_matrixGPU, MatToTrans->getNCol());
            break;
        case 128:
            multiplyGPUMatVectTrans<128> << <numBlock, _blockSize >> > (_matrixGPU, MatToTrans->_matrixGPU, vect->_matrixGPU, MatToTrans->getNCol());
            break;
        case 64:
            multiplyGPUMatVectTrans< 64> << <numBlock, _blockSize >> > (_matrixGPU, MatToTrans->_matrixGPU, vect->_matrixGPU, MatToTrans->getNCol());
            break;
        case 32:
            multiplyGPUMatVectTrans< 32> << <numBlock, _blockSize >> > (_matrixGPU, MatToTrans->_matrixGPU, vect->_matrixGPU, MatToTrans->getNCol());
            break;
        case 16:
            multiplyGPUMatVectTrans< 16> << <numBlock, _blockSize >> > (_matrixGPU, MatToTrans->_matrixGPU, vect->_matrixGPU, MatToTrans->getNCol());
            break;
        case  8:
            multiplyGPUMatVectTrans<  8> << <numBlock, _blockSize >> > (_matrixGPU, MatToTrans->_matrixGPU, vect->_matrixGPU, MatToTrans->getNCol());
            break;
        case  4:
            multiplyGPUMatVectTrans<  4> << <numBlock, _blockSize >> > (_matrixGPU, MatToTrans->_matrixGPU, vect->_matrixGPU, MatToTrans->getNCol());
            break;
        case  2:
            multiplyGPUMatVectTrans<  2> << <numBlock, _blockSize >> > (_matrixGPU, MatToTrans->_matrixGPU, vect->_matrixGPU, MatToTrans->getNCol());
            break;
        case  1:
            multiplyGPUMatVectTrans<  1> << <numBlock, _blockSize >> > (_matrixGPU, MatToTrans->_matrixGPU, vect->_matrixGPU, MatToTrans->getNCol());
            break;
        }

    }
    else if (!_GPU && !(MatToTrans->getPos()) && !vect->getPos())
    {
        if (rowVector) {
            for (int i = 0; i < MatToTrans->getNLin(); ++i)
            {
                float sum = 0;
                for (int j = 0; j < MatToTrans->getNCol(); ++j)
                {
                    sum += MatToTrans->get(j, i) * vect->get(0, j);
                }
                set(0, i, sum);
            }
        }
        else {
            for (int i = 0; i < _row; ++i)
            {
                float sum = 0;
                for (int j = 0; j < MatToTrans->getNCol(); ++j)
                {
                    sum += MatToTrans->get(j, i) * vect->get(j, 0);
                }
                set(i, 0, sum);
            }
        }
    }
    else {
        throw std::invalid_argument("MultiplyMatTransVec Matrix not at the same place");
    }

}

void MatrixGPU::linearOperation(MatrixGPU* A, MatrixGPU* x, MatrixGPU* b, bool trans)
{
    if (trans) {
        if (getNLin() != 1) {
            throw std::invalid_argument("linearOperation result must be a row vector ");
        }
        else if (getNCol() != A->getNLin()) {
            throw std::invalid_argument("linearOperation result must be compatible with A");
        }
        else if (x->getNLin() != 1 || b->getNLin() != 1) {
            throw std::invalid_argument("linearOperation x and b must be a row vector ");
        }
        else if (x->getNCol() != A->getNCol()) {
            throw std::invalid_argument("linearOperation x must be compatible with A");
        }
    }
    else {
        if (getNCol() != 1) {
            throw std::invalid_argument("linearOperation result must be a column vector ");
        }
        else if (getNLin() != A->getNLin()) {
            throw std::invalid_argument("linearOperation result must have the same row number as A");
        }
        else if (x->getNCol() != 1 || b->getNCol() != 1) {
            throw std::invalid_argument("linearOperation x and b must be a column vector ");
        }
        else if (x->getNLin() != A->getNCol()) {
            throw std::invalid_argument("linearOperation x must be compatible with Mat");
        }
    }

    if (_GPU && A->getPos() && b->getPos() && x->getPos())
    {
        int numBlock = A->getNLin();
        switch (_blockSize) {
        case 512:
            linearOpGPU<512> << <numBlock, _blockSize >> > (_matrixGPU, A->_matrixGPU, x->_matrixGPU, b->_matrixGPU, A->getNCol());
            break;
        case 256:
            linearOpGPU<256> << <numBlock, _blockSize >> > (_matrixGPU, A->_matrixGPU, x->_matrixGPU, b->_matrixGPU, A->getNCol());
            break;
        case 128:
            linearOpGPU<128> << <numBlock, _blockSize >> > (_matrixGPU, A->_matrixGPU, x->_matrixGPU, b->_matrixGPU, A->getNCol());
            break;
        case 64:
            linearOpGPU< 64> << <numBlock, _blockSize >> > (_matrixGPU, A->_matrixGPU, x->_matrixGPU, b->_matrixGPU, A->getNCol());
            break;
        case 32:
            linearOpGPU< 32> << <numBlock, _blockSize >> > (_matrixGPU, A->_matrixGPU, x->_matrixGPU, b->_matrixGPU, A->getNCol());
            break;
        case 16:
            linearOpGPU< 16> << <numBlock, _blockSize >> > (_matrixGPU, A->_matrixGPU, x->_matrixGPU, b->_matrixGPU, A->getNCol());
            break;
        case  8:
            linearOpGPU<  8> << <numBlock, _blockSize >> > (_matrixGPU, A->_matrixGPU, x->_matrixGPU, b->_matrixGPU, A->getNCol());
            break;
        case  4:
            linearOpGPU<  4> << <numBlock, _blockSize >> > (_matrixGPU, A->_matrixGPU, x->_matrixGPU, b->_matrixGPU, A->getNCol());
            break;
        case  2:
            linearOpGPU<  2> << <numBlock, _blockSize >> > (_matrixGPU, A->_matrixGPU, x->_matrixGPU, b->_matrixGPU, A->getNCol());
            break;
        case  1:
            linearOpGPU<  1> << <numBlock, _blockSize >> > (_matrixGPU, A->_matrixGPU, x->_matrixGPU, b->_matrixGPU, A->getNCol());
            break;
        }

    }
    else if (!_GPU && !(A->getPos()) && !x->getPos() && !b->getPos())
    {
        if (trans) {
            for (int i = 0; i < A->getNLin(); ++i)
            {
                float sum = 0;
                for (int j = 0; j < A->getNCol(); ++j)
                {
                    sum += A->get(i, j) * x->get(0, j);
                }
                set(0, i, sum + b->get(0,i));
            }
        }
        else {
            for (int i = 0; i < _row; ++i)
            {
                float sum = 0;
                for (int j = 0; j < A->getNCol(); ++j)
                {
                    sum += A->get(i, j) * x->get(j, 0);
                }
                set(i, 0, sum + b->get(i,0));
            }
        }
    }
    else {
        throw std::invalid_argument("linearOperation Matrix not at the same place");
    }
}

///////////////////////////////////////////////////////////////////////////////
// Multiplication Terme � Terme
///////////////////////////////////////////////////////////////////////////////

void MatrixGPU::multiplyT(MatrixGPU* m)
{
    if (!dim(m)) {
        throw std::invalid_argument("multiplyT not the same dimension");
    }
   
    if (_GPU && m->getPos())
    {
        if (_N > 0) {
            multiplyTGPU << <_numBlocks, _blockSize >> > (_matrixGPU, m->_matrixGPU, _N);
        }
    }
    else if (!_GPU && !(m->getPos()))
    {
        for (int i = 0;i < _row;++i)
        {
            for (int j = 0;j < _column;++j)
            {
                float r = get(i, j) * m->get(i, j);
                this->set(i, j, r);
            }
        }
    }
    else {
        throw std::invalid_argument("multiplyT Matrix not at the same place");
    }
}

void MatrixGPU::multiplyT(MatrixGPU* m1, MatrixGPU* m2)
{
    if (!m1->dim(m2)) {
        throw std::invalid_argument("multiplyT not the same dimension");
    }
    if (!dim(m1)) {
        throw std::invalid_argument("multiplyT not the same dimension");
    }
    if (_GPU && m1->getPos() && m2->getPos())
    {
        multiplyTGPU<<<_numBlocks, _blockSize >>>(_matrixGPU, m1->_matrixGPU, m2->_matrixGPU, _N);
    }
    else if (!_GPU && !(m1->getPos()) && !(m2->getPos()))
    {
        for (int i = 0;i < _row;++i)
        {
            for (int j = 0;j < _column;++j)
            {
                float r = m1->get(i, j) * m2->get(i, j);
                this->set(i, j, r);
            }
        }
    }
    else {
        throw std::invalid_argument("multiplyT Matrix not at the same place");
    }
}


void MatrixGPU::divide(float c)
{
    if (c == 0) {
        throw std::domain_error("divide : divide by 0");
    }
    if (_GPU) {
        divideGPU <<<_numBlocks, _blockSize >>> (_matrixGPU, c, _N);
    }
    else {
        for (int i = 0;i < _row;++i)
        {
            for (int j = 0;j < _column;++j)
            {
                float r = get(i, j) / c;
                this->set(i, j, r);
            }
        }
    }
    
}

void MatrixGPU::divideT(MatrixGPU* m)
{
    
    if (!dim(m)) {
        throw std::invalid_argument("divideT not the same dimension");
    }
    if (_GPU && m->getPos())
    {
        divideGPU<<<_numBlocks, _blockSize >>>(_matrixGPU, m->_matrixGPU, _N);
    }
    else if (!_GPU && !(m->getPos()))
    {
        MatrixGPU temp(*this);
        float r = 0;
        float f = 0;
        for (int i = 0;i < _row;++i)
        {
            for (int j = 0;j < _column;++j)
            {
                f = m->get(i, j);
                if (f == 0) {
                    throw std::domain_error("divideT divide by 0");
                }
                r = get(i, j) / f;
                temp.set(i, j, r);
            }
        }
        set(&temp);
    }
    else {
        throw std::invalid_argument("divideT Matrix not at the same place");
    }
    
}

void MatrixGPU::invertGaussJordan(MatrixGPU* mToInvert)
{
    
    if (!dim(mToInvert)) {
        throw std::invalid_argument("invertGaussJordan not the same dimension");
    }
    if (_row != _column) {
        throw std::invalid_argument("invertGaussJordan must be a square matrix");
    }
   
    if (!_GPU && !mToInvert->getPos()) {
        MatrixCPU m;
        mToInvert->toMatCPU(m);
       
        MatrixCPU augmented(_row, _column);
        augmented.setEyes(1);
        MatrixCPU indices(1, 2);
        int r = 0;
        for (int column = 0; column < _column; column++) {
            
            float pivotabs = m.maxAbs(r, _row, column, column + 1, &indices);
            int k = indices.get(0, 0); // indice max de la colonne j
            float pivot = m.get(k, column);
            if (pivotabs < 0.000001f) {
                throw std::invalid_argument("invertGaussJordan not invertible matrix");
            }
            else {
                
                for (int j = 0; j < _column; j++) {
                    augmented.set(k, j, augmented.get(k, j) / pivot);
                    m.set(k, j, m.get(k, j) / pivot);
                }
                
                if (k != r) {
                    augmented.swapLine(k, r);
                    m.swapLine(k, r);
                }
                
                for (int i = 0; i < _row; i++) {
                    if (i != r) {
                        float local = m.get(i, column);
                        m.subtractRow(i, r, local);
                        augmented.subtractRow(i, r, local);
                    }
                }
                r++;
            }
        }
        
        set(&augmented);
       
    }
   
    else if (_GPU && mToInvert->getPos()) {
        MatrixGPU m(*mToInvert);
        setEyes(1);
        dim3 threadsPerBlock(32, 32);
        int r = 0;
        MatrixGPU matCol(_row, 1, 0, 1);
        int k = 0;
        
        for (int column = 0; column < _column; column++) {
           
            //transferCPU();
            ///display();
            //transferGPU();

            m.getCol(&matCol, column, r);
            float pivotAbs = matCol.max2(&k); // comme matCol est un vecteur colonne, la position du maximum correspond directement à la ligne !
            float pivot = matCol.get(k, 0, false);
            
            
            if (pivotAbs < 0.000001f) {
                std::cout << "not invertible " << column <<" " << pivotAbs << std::endl;
                if (_N < 100) {
                    mToInvert->display(true);
                }
                throw std::invalid_argument("invertGaussJordan not invertible matrix");
            }
            else {
                normalisationGJ << <_numBlocks, _blockSize >> > (_matrixGPU, k, _column, pivot); // normalisation pour toute la ligne k
                normalisationGJ <<<_numBlocks, _blockSize >>> (m._matrixGPU, k, _column, pivot);
                
              
                if (k != r) {
                    swapLineGJ << <_numBlocks, _blockSize >> > (_matrixGPU, k, r, _column);// swap des lignes k et r 
                    swapLineGJ << <_numBlocks, _blockSize >> > (m._matrixGPU, k, r, _column);// swap des lignes k et r 
                }
                
                // soustration de la ligne sauf pour la r
                eliminationGJ <<<_row, _blockSize >> > (m._matrixGPU, _matrixGPU, r, _row, _column);
                r++;
            }
        }
    }
    else {
        throw std::invalid_argument("invertGaussJordan Matrix not at the same place");
    }


}

void MatrixGPU::LUPFactorization(MatrixGPU* A, MatrixGPU* P)
{
    float Tol = 0.0000001;
    int n = getNLin();
    A->set(this);

    // code from wikipedia adapted
    if (getNCol() != getNLin()) {
        throw std::invalid_argument("LUPFactorization A must be square");
    }
    if (P->getNCol() != 1 || P->getNLin() != (getNCol() + 1)) {
        throw std::invalid_argument("LUPFactorization wrong size of P");
    }

    if (!_GPU && !A->getPos() && !P->getPos()) {
        for (int i = 0; i < n; i++) {
            P->set(i, 0, i); //Unit permutation matrix, P[N] initialized with N
        }

        float absA = 0;
        int j = 0;
        for (int col = 0; col < n; col++) {
            float maxA = 0.0;
            int imax = col;
            for (int k = col; k < n; k++) {
                absA = fabs(A->get(k, col));
                if (absA > maxA)
                {
                    maxA = absA;
                    imax = k;
                }
            }
            //std::cout << "max de " << maxA << "en position " << imax << std::endl;
            if (maxA < Tol) {
                //std::cout << "failure, matrix is degenerate" << std::endl;
                throw std::invalid_argument("LUPFactorization matrix is degenerate");
                return; //failure, matrix is degenerate
            }

            if (imax != col) { //le max pas sur la diagonal
                //pivoting P
                j = P->get(col, 0);
                P->set(col, 0, P->get(imax, 0));
                P->set(imax, 0, j);

                //pivoting rows of A
               
                A->swapLine(col, imax);
               
                //counting pivots starting from N (for determinant)
                P->set(n, 0, P->get(n, 0) + 1);
            }


            for (int i = col + 1; i < n; i++) {

                A->set(i, col, A->get(i, col) / A->get(col, col)); //A[j][i] /= A[i][i];


                for (int k = col + 1; k < n; k++) {
                    A->set(i, k, A->get(i, k) - A->get(i, col) * A->get(col, k)); //A[j][k] -= A[j][i] * A[i][k];

                }


            }
        }
    } 
    else if (_GPU && A->getPos() && P->getPos()) {

        MatrixGPU matCol(_row, 1, 0, 1);
        initPermMatr <<<_numBlocks, _blockSize >> > (P->_matrixGPU, n);
       
        int k = 0;
        for (int col = 0; col < n; col++) {

            A->getCol(&matCol, col, col);
            float pivotAbs = matCol.max2(&k); // comme matCol est un vecteur colonne, la position du maximum correspond directement à la ligne !
            float pivot = matCol.get(k, 0, false);

            //std::cout << "max de " << pivot << "en position " << k << std::endl;

            if (pivotAbs < Tol) {
                //std::cout << "failure, matrix is degenerate" << std::endl;
             
                throw std::invalid_argument("LUPFactorization matrix is degenerate");
            }
            else {
                if (k != col) { //le max pas sur la diagonal
                    //pivoting P and counting pivots starting from N (for determinant)
                    updatePermMatr <<<1, 1 >>> (P->_matrixGPU, k, col, n);
                 
                    //pivoting rows of A
                    A->swapLine(col, k);
                }
                
                updateLUPFactorization << <n, _blockSize >> > (A->_matrixGPU, col, n);
            }
        }
    }
    else {
        throw std::invalid_argument("LUPFactorization Matrix not at the same place");
    }
    
    // en vrai on peut tout stocker dans une matrice comme on sait que diag(L) = Id, Et donc on peut avoir A = (L-Id) + U


}

void MatrixGPU::solveSysUpper(MatrixGPU* U)
{
    if (getNLin() != U->getNCol() || U->getNLin() != getNLin()) {
        throw std::invalid_argument("solveSysUpper A must be square");
    }
    int n = getNLin();
    if (!_GPU && !U->getPos()) {
    
        for (int i = n - 1; i >= 0; i--)
        {
            for (int k = i + 1; k < n; k++) {
                set(i, 0, get(i, 0) - U->get(i, k) * get(k, 0));// x[i] -= A[i][k] * x[k];
            }

            set(i, 0, get(i, 0) / U->get(i, i));
        }
    }
    else if (_GPU && U->getPos()) {
        for (int i = n - 1; i >= 0; i--)
        {
            solveUpSys << < 1, _blockSize >> > (U->_matrixGPU, _matrixGPU, i, n);
        }
    
    }
    else {
        throw std::invalid_argument("solveSysUpper Matrix not at the same place");
    }
}

void MatrixGPU::solveSysLower(MatrixGPU* L, MatrixGPU* b, MatrixGPU* P) // element diag equal to 1
{
    if (getNLin() != L->getNCol() || L->getNLin() != b->getNLin()) {
        throw std::invalid_argument("solveSysLower A must be square");
    }
    int n = getNLin();
    if (!_GPU && !L->getPos() && !P->getPos() && !b->getPos()) {
        for (int i = 0; i < n; i++) {
            set(i, 0, b->get(P->get(i, 0), 0)); // x[i] = b[P[i]];

            for (int k = 0; k < i; k++) {
                set(i, 0, get(i, 0) - L->get(i, k) * get(k, 0));
            }
        }
    }
    else if (_GPU && L->getPos() && P->getPos() && b->getPos()) {
        setPermute << <_numBlocks, _blockSize >> > (_matrixGPU, b->_matrixGPU, P->_matrixGPU, n);
        for (int i = 0; i < n; i++) {
            solveLowSys << < 1, _blockSize >> > (L->_matrixGPU, _matrixGPU, i, n);
        }
    }
    else {
        throw std::invalid_argument("solveSysLower Matrix not at the same place");
    }
    
}

void MatrixGPU::solveSys(MatrixGPU* A, MatrixGPU* P, MatrixGPU* b)
{
    if (A->getNCol() != A->getNLin() || A->getNLin() != b->getNLin()) {
        throw std::invalid_argument("solveSys wrong size of A");
    }
    if (b->getNCol() != 1) {
        throw std::invalid_argument("solveSys b must be a column vector");
    }
    if (P->getNLin() != (A->getNCol() + 1) || P->getNCol() != 1) {
        throw std::invalid_argument("solveSys wrong size of P");
    }

    if (!_GPU && !A->getPos() && !P->getPos() && !b->getPos()) {
        int n = getNLin();
        for (int i = 0; i < n; i++) {
            set(i, 0, b->get(P->get(i, 0), 0)); // x[i] = b[P[i]];

            for (int k = 0; k < i; k++) {
                set(i, 0, get(i, 0) - A->get(i, k) * get(k, 0));
            }
        }
        for (int i = n - 1; i >= 0; i--)
        {
            for (int k = i + 1; k < n; k++) {
                set(i, 0, get(i, 0) - A->get(i, k) * get(k, 0));
               
            }

            set(i, 0, get(i, 0) / A->get(i, i));
        }
    }
    else if (_GPU && A->getPos() && P->getPos() && b->getPos()) {
        int n = getNLin();
        setPermute << <_numBlocks, _blockSize >> > (_matrixGPU, b->_matrixGPU, P->_matrixGPU, n);
        
        //CHECK_CUDA_ERROR_MAT(hipFuncSetAttribute(reinterpret_cast<const void*>(solveSysGPU), hipFuncAttributeMaxDynamicSharedMemorySize, n * sizeof(float)));

        solveSysGPU<<<1, _blockSize, n*sizeof(float)>>> (A->_matrixGPU, _matrixGPU, n);
        
        
        ////CHECK_LAST_CUDA_ERROR_MAT();

        //CHECK_CUDA_ERROR_MAT(hipDeviceSynchronize());/**/
        /*for (int i = 0; i < n; i++) {
            solveLowSys <<< 1, _blockSize >>> (A->_matrixGPU, _matrixGPU, i, n);
        }
       
        for (int i = n - 1; i >= 0; i--)
        {
            solveUpSys << < 1, _blockSize >> > (A->_matrixGPU, _matrixGPU, i, n);
        }*/
    }
    else {
        throw std::invalid_argument("solveSys Matrix not at the same place");
    }
    
   
}

///////////////////////////////////////////////////////////////////////////////
// Fonction autres
///////////////////////////////////////////////////////////////////////////////

float MatrixGPU::max2() const
{
    if (_row == 0 || _column == 0) {
        return 0;
        //throw std::out_of_range("Empty Matrix");
    }
    if (!_GPU) {
        float M = fabs(get(0, 0));
        float m = 0;
        for (int i = 0;i < _row;++i)
        {
            for (int j = 0;j < _column;++j)
            {
                m = fabs(get(i, j));
                if (m > M) {
                    M = m;
                }
            }
        }
        return M;
    } 
    else {    
        int numBlocks = _numBlocks;
        unsigned int n = _N;
        float odata = 0;
        float* d_odata;
        if (preallocation) {
            d_odata = _preallocation;
        }
        else {
            std::cout << "allocation !!!" << std::endl;
            hipMalloc((void**)&d_odata, sizeof(float) * numBlocks);
        }
        //std::cout << _numBlocks << std::endl;

        switch (_blockSize) {
        case 512:
            maxMultiBlock<512> << <numBlocks, _blockSize >> > (_matrixGPU, d_odata, n);
            maxMonoBlock<512> << < 1, _blockSize >> > (d_odata, d_odata, numBlocks);
            break;
        case 256:
            maxMultiBlock<256> << <numBlocks, _blockSize >> > (_matrixGPU, d_odata, n);
            maxMonoBlock<256> << < 1, _blockSize >> > (d_odata, d_odata, numBlocks);
            break;
        case 128:
            maxMultiBlock<128> << <numBlocks, _blockSize >> > (_matrixGPU, d_odata, n);
            maxMonoBlock<128> << < 1, _blockSize >> > (d_odata, d_odata, numBlocks);
            break;
        case 64:
            maxMultiBlock< 64> << <numBlocks, _blockSize >> > (_matrixGPU, d_odata, n);
            maxMonoBlock< 64> << < 1, _blockSize >> > (d_odata, d_odata, numBlocks);
            break;
        case 32:
            maxMultiBlock< 32> << <numBlocks, _blockSize >> > (_matrixGPU, d_odata, n);
            maxMonoBlock< 32> << < 1, _blockSize >> > (d_odata, d_odata, numBlocks);
            break;
        case 16:
            maxMultiBlock< 16> << <numBlocks, _blockSize >> > (_matrixGPU, d_odata, n);
            maxMonoBlock< 16> << < 1, _blockSize >> > (d_odata, d_odata, numBlocks);
            break;
        case  8:
            maxMultiBlock<  8> << <numBlocks, _blockSize >> > (_matrixGPU, d_odata, n);
            maxMonoBlock< 8> << < 1, _blockSize >> > (d_odata, d_odata, numBlocks);
            break;
        case  4:
            maxMultiBlock<  4> << <numBlocks, _blockSize >> > (_matrixGPU, d_odata, n);
            maxMonoBlock<  4> << < 1, _blockSize >> > (d_odata, d_odata, numBlocks);
            break;
        case  2:
            maxMultiBlock<  2> << <numBlocks, _blockSize >> > (_matrixGPU, d_odata, n);
            maxMonoBlock<  2> << < 1, _blockSize >> > (d_odata, d_odata, numBlocks);
            break;
        case  1:
            maxMultiBlock<  1> << <numBlocks, _blockSize >> > (_matrixGPU, d_odata, n);
            maxMonoBlock<  1> << < 1, _blockSize >> > (d_odata, d_odata, numBlocks);
            break;
        }
        //hipDeviceSynchronize();
         if (preallocation) {
             hipMemcpy(&odata, d_odata, sizeof(float), hipMemcpyDeviceToHost);
             //hipMemcpy(_preallocationFloat, d_odata, sizeof(float), hipMemcpyDeviceToHost);
             return sqrt(odata);//sqrt(*_preallocationFloat);
        }
        else
        {
            hipMemcpy(&odata, d_odata, sizeof(float), hipMemcpyDeviceToHost);
            std::cout << "free !!!" << std::endl;
            hipFree(d_odata);
            return sqrt(odata);
        }
    }
}

float MatrixGPU::max2(int* indice)
{
    if (_row == 0 || _column == 0) {
        throw std::out_of_range("max2 Empty Matrix");
    }
    if (!_GPU) {
        float M = fabs(get(0, 0));
        float m = 0;
        for (int i = 0; i < _row; ++i)
        {
            for (int j = 0; j < _column; ++j)
            {
                m = fabs(get(i, j));
                if (m > M) {
                    *indice = i * _column + j;
                    M = m;
                }
            }
        }
        return M;
    }
    else {

        int numBlocks = _numBlocks;
        unsigned int n = _N;
        float odata = 0;
        float* d_odata;
        int* d_pos;
        int pos = 0;
        hipMalloc((void**)&d_pos, sizeof(int) * numBlocks);
        if (preallocation) {

            d_odata = _preallocation;
        }
        else {

            hipMalloc((void**)&d_odata, sizeof(float) * numBlocks);
        }
        //std::cout << _numBlocks << std::endl;

        switch (_blockSize) {
        case 512:
            maxMultiBlock<512> <<<numBlocks, _blockSize >> > (_matrixGPU, d_odata, n, d_pos);
            maxMonoBlock<512> <<< 1, _blockSize >> > (d_odata, d_odata, numBlocks, d_pos);
            break;
        case 256:
            maxMultiBlock<256> << <numBlocks, _blockSize >> > (_matrixGPU, d_odata, n, d_pos);
            maxMonoBlock<256> << < 1, _blockSize >> > (d_odata, d_odata, numBlocks, d_pos);
            break;
        case 128:
            maxMultiBlock<128> << <numBlocks, _blockSize >> > (_matrixGPU, d_odata, n, d_pos);
            maxMonoBlock<128> << < 1, _blockSize >> > (d_odata, d_odata, numBlocks, d_pos);
            break;
        case 64:
            maxMultiBlock< 64> << <numBlocks, _blockSize >> > (_matrixGPU, d_odata, n, d_pos);
            maxMonoBlock< 64> << < 1, _blockSize >> > (d_odata, d_odata, numBlocks, d_pos);
            break;
        case 32:
            maxMultiBlock< 32> << <numBlocks, _blockSize >> > (_matrixGPU, d_odata, n, d_pos);
            maxMonoBlock< 32> << < 1, _blockSize >> > (d_odata, d_odata, numBlocks, d_pos);
            break;
        case 16:
            maxMultiBlock< 16> << <numBlocks, _blockSize >> > (_matrixGPU, d_odata, n, d_pos);
            maxMonoBlock< 16> << < 1, _blockSize >> > (d_odata, d_odata, numBlocks, d_pos);
            break;
        case  8:
            maxMultiBlock<  8> << <numBlocks, _blockSize >> > (_matrixGPU, d_odata, n, d_pos);
            maxMonoBlock< 8> << < 1, _blockSize >> > (d_odata, d_odata, numBlocks, d_pos);
            break;
        case  4:
            maxMultiBlock<  4> << <numBlocks, _blockSize >> > (_matrixGPU, d_odata, n, d_pos);
            maxMonoBlock<  4> << < 1, _blockSize >> > (d_odata, d_odata, numBlocks, d_pos);
            break;
        case  2:
            maxMultiBlock<  2> << <numBlocks, _blockSize >> > (_matrixGPU, d_odata, n, d_pos);
            maxMonoBlock<  2> << < 1, _blockSize >> > (d_odata, d_odata, numBlocks, d_pos);
            break;
        case  1:
            maxMultiBlock<  1> << <numBlocks, _blockSize >> > (_matrixGPU, d_odata, n, d_pos);
            maxMonoBlock<  1> << < 1, _blockSize >> > (d_odata, d_odata, numBlocks, d_pos);
            break;
        }
        //hipDeviceSynchronize();
        hipMemcpy(&odata, d_odata, sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(&pos, d_pos, sizeof(int), hipMemcpyDeviceToHost);
        hipFree(d_pos);
        if (!preallocation) {
            hipFree(d_odata);
        }
        *indice = pos;
        return sqrt(odata);
    }
}
float MatrixGPU::max2(MatrixGPU* m) const
{
    if (_row == 0 || _column == 0) {
        throw std::out_of_range("max2 Empty Matrix");
    }
    if (!_GPU && !(m->getPos())) 
    {
        float M = 0;
        float f = 0;
        for (int i = 0;i < _row;++i)
        {
            for (int j = 0;j < _column;++j)
            {
                f = fabs(get(i, j)- m->get(i,j));
                if (f > M) {
                    M = f;
                }
            }
        }
        return M;
    }
    else if (_GPU && m->getPos()) {
        int numBlocks = _numBlocks;
        unsigned int n = _N;
        float odata;
        float* d_odata;
        if (preallocation) {
            d_odata = _preallocation;
        }
        else {
            std::cout << "allocation !!!" << std::endl;
            hipMalloc((void**)&d_odata, sizeof(float) * numBlocks);
        }
        //std::cout << _numBlocks << std::endl;

        switch (_blockSize) {
        case 512:
            maxMultiBlock<512> << <numBlocks, _blockSize >> > (_matrixGPU, m->_matrixGPU, d_odata, n);
            maxMonoBlock<512> << < 1, _blockSize >> > (d_odata, d_odata, numBlocks);
            break;
        case 256:
            maxMultiBlock<256> << <numBlocks, _blockSize >> > (_matrixGPU, m->_matrixGPU, d_odata, n);
            maxMonoBlock<256> << < 1, _blockSize >> > (d_odata, d_odata, numBlocks);
            break;
        case 128:
            maxMultiBlock<128> << <numBlocks, _blockSize >> > (_matrixGPU, m->_matrixGPU, d_odata, n);
            maxMonoBlock<128> << < 1, _blockSize >> > (d_odata, d_odata, numBlocks);
            break;
        case 64:
            maxMultiBlock< 64> << <numBlocks, _blockSize >> > (_matrixGPU, m->_matrixGPU, d_odata, n);
            maxMonoBlock< 64> << < 1, _blockSize >> > (d_odata, d_odata, numBlocks);
            break;
        case 32:
            maxMultiBlock< 32> << <numBlocks, _blockSize >> > (_matrixGPU, m->_matrixGPU, d_odata, n);
            maxMonoBlock< 32> << < 1, _blockSize >> > (d_odata, d_odata, numBlocks);
            break;
        case 16:
            maxMultiBlock< 16> << <numBlocks, _blockSize >> > (_matrixGPU, m->_matrixGPU, d_odata, n);
            maxMonoBlock< 16> << < 1, _blockSize >> > (d_odata, d_odata, numBlocks);
            break;
        case  8:
            maxMultiBlock<  8> << <numBlocks, _blockSize >> > (_matrixGPU, m->_matrixGPU, d_odata, n);
            maxMonoBlock< 8> << < 1, _blockSize >> > (d_odata, d_odata, numBlocks);
            break;
        case  4:
            maxMultiBlock<  4> << <numBlocks, _blockSize >> > (_matrixGPU, m->_matrixGPU, d_odata, n);
            maxMonoBlock<  4> << < 1, _blockSize >> > (d_odata, d_odata, numBlocks);
            break;
        case  2:
            maxMultiBlock<  2> << <numBlocks, _blockSize >> > (_matrixGPU, m->_matrixGPU, d_odata, n);
            maxMonoBlock<  2> << < 1, _blockSize >> > (d_odata, d_odata, numBlocks);
            break;
        case  1:
            maxMultiBlock<  1> << <numBlocks, _blockSize >> > (_matrixGPU, m->_matrixGPU, d_odata, n);
            maxMonoBlock<  1> << < 1, _blockSize >> > (d_odata, d_odata, numBlocks);
            break;
        }
        //hipDeviceSynchronize();
        if (preallocation) {
            hipMemcpy(&odata, d_odata, sizeof(float), hipMemcpyDeviceToHost);
            //hipMemcpy(_preallocationFloat, d_odata, sizeof(float), hipMemcpyDeviceToHost);
            return  sqrt(odata);//sqrt(*_preallocationFloat);
        }
        else
        {
            hipMemcpy(&odata, d_odata, sizeof(float), hipMemcpyDeviceToHost);
            std::cout << "free !!!" << std::endl;
            hipFree(d_odata);
            return sqrt(odata);
        }
    }
    else {
        throw std::invalid_argument("max2 Matrix not at the same place");
    }
}

float MatrixGPU::distance2(MatrixGPU* m)
{
    if (!dim(m)) {
        throw std::invalid_argument("distance2 not the same size");
    }
    if (_GPU && m->getPos())
    {
        int numBlocks = _numBlocks;
        unsigned int n = _N;
        float odata = 0;
        float* d_odata;
        if (preallocation) {
            d_odata = _preallocation;
        }
        else {
            hipMalloc((void**)&d_odata, sizeof(float) * numBlocks);
        }
        //std::cout << _numBlocks << std::endl;

        switch (_blockSize) {
        case 512:
            distanceMultiBlock<512> << <numBlocks, _blockSize >> > (_matrixGPU, m->_matrixGPU, d_odata, n);
            sumMonoBlock<512> << < 1, _blockSize >> > (d_odata, d_odata, numBlocks);
            break;
        case 256:
            distanceMultiBlock<256> << <numBlocks, _blockSize >> > (_matrixGPU, m->_matrixGPU, d_odata, n);
            sumMonoBlock<256> << < 1, _blockSize >> > (d_odata, d_odata, numBlocks);
            break;
        case 128:
            distanceMultiBlock<128> << <numBlocks, _blockSize >> > (_matrixGPU, m->_matrixGPU, d_odata, n);
            sumMonoBlock<128> << < 1, _blockSize >> > (d_odata, d_odata, numBlocks);
            break;
        case 64:
            distanceMultiBlock< 64> << <numBlocks, _blockSize >> > (_matrixGPU, m->_matrixGPU, d_odata, n);
            sumMonoBlock< 64> << < 1, _blockSize >> > (d_odata, d_odata, numBlocks);
            break;
        case 32:
            distanceMultiBlock< 32> << <numBlocks, _blockSize >> > (_matrixGPU, m->_matrixGPU, d_odata, n);
            sumMonoBlock< 32> << < 1, _blockSize >> > (d_odata, d_odata, numBlocks);
            break;
        case 16:
            distanceMultiBlock< 16> << <numBlocks, _blockSize >> > (_matrixGPU, m->_matrixGPU, d_odata, n);
            sumMonoBlock< 16> << < 1, _blockSize >> > (d_odata, d_odata, numBlocks);
            break;
        case  8:
            distanceMultiBlock<  8> << <numBlocks, _blockSize >> > (_matrixGPU, m->_matrixGPU, d_odata, n);
            sumMonoBlock< 8> << < 1, _blockSize >> > (d_odata, d_odata, numBlocks);
            break;
        case  4:
            distanceMultiBlock<  4> << <numBlocks, _blockSize >> > (_matrixGPU, m->_matrixGPU, d_odata, n);
            sumMonoBlock<  4> << < 1, _blockSize >> > (d_odata, d_odata, numBlocks);
            break;
        case  2:
            distanceMultiBlock<  2> << <numBlocks, _blockSize >> > (_matrixGPU, m->_matrixGPU, d_odata, n);
            sumMonoBlock<  2> << < 1, _blockSize >> > (d_odata, d_odata, numBlocks);
            break;
        case  1:
            distanceMultiBlock<  1> << <numBlocks, _blockSize >> > (_matrixGPU, m->_matrixGPU, d_odata, n);
            sumMonoBlock<  1> << < 1, _blockSize >> > (d_odata, d_odata, numBlocks);
            break;
        }
        //hipDeviceSynchronize();
        hipMemcpy(&odata, d_odata, sizeof(float), hipMemcpyDeviceToHost);
        if (!preallocation) {
            hipFree(d_odata);
        }


        return sqrtf(odata);
    }
    else if (!_GPU && !(m->getPos()))
    {
        double d = 0;
        double r = 0;
        for (int i = 0; i < _row;++i)
        {
            for (int j = 0; j < _column;++j)
            {
                r = get(i, j) - m->get(i, j);
                d = d + r * r;
            }
        }
        return sqrtf(d);
    }
    else {
        throw std::invalid_argument("distance2 Matrix not at the same place");
    }
}

void MatrixGPU::Moy(MatrixGPU* m, MatrixGPU* nb, int sens)
{
    float s;
    int n;
    if (sens) { // on travaille sur les colonnes
        if ((_row != 1) || (_column != m->getNCol()) || (_column != nb->getNCol()) || (nb->getNLin() != 1))
        {
            throw std::invalid_argument("Moy wrong dimension of the vector");
        }
        if (_GPU && nb->getPos() && m->getPos())
        {
            
            moyGPU1 <<<_numBlocks, _blockSize >>> (_matrixGPU, m->_matrixGPU, nb->_matrixGPU, m->getNLin(), _column);
        }
        else if ((!_GPU) && !(nb->getPos()) && !(m->getPos())) {
            for (int j = 0; j < _column;j++)
            {
                n = nb->get(0, j);
                s = 0;
                if (n > 0)
                {
                    for (int i = 0; i < m->getNLin();i++)
                    {
                        s = s + m->get(i, j);
                    }
                    s = s / n;
                }
                set(0, j, s);
            }
        }
        else {
            throw std::invalid_argument("Moy Matrix not at the same place");
        }
        

    }
    else { // on travaille sur les lignes 
        if ((_column != 1) || (_row != m->getNLin()) || (_row != nb->getNLin()) || (nb->getNCol() != 1)) {
            throw std::invalid_argument("Moy wrong dimension of the vector");
        }
        if (_GPU && nb->getPos())
        {
            
            moyGPU2<<<_numBlocks, _blockSize >> > (_matrixGPU, m->_matrixGPU, nb->_matrixGPU, _row , m->getNCol());
        }
        else if ((!_GPU) && !(nb->getPos())) {
            for (int i = 0; i < _row;i++)
            {
                n = nb->get(i, 0);
                s = 0;
                if (n > 0) {
                    for (int j = 0; j < m->getNCol();j++)
                    {
                        s = s + m->get(i, j);
                    }
                    s = s / n;
                }
                set(i, 0, s);
            }
        }
        else {
            throw std::invalid_argument("Moy Matrix not at the same place");
        }

    }
}

void MatrixGPU::project(MatrixGPU* Lb, MatrixGPU* Ub)
{
    if (!dim(Lb) || !dim(Ub)) {
        throw std::invalid_argument("project not the same dimension");
    }
    if (_GPU && Lb->getPos() && Ub->getPos())
    {
        projectGPU<<<_numBlocks, _blockSize >>>(_matrixGPU, Lb->_matrixGPU, Ub->_matrixGPU, _N);
    }
    else if (!_GPU && !(Lb->getPos()) && !(Ub->getPos()))
    {
        float ub = 0;
        float lb = 0;
        float r = 0;
        MatrixGPU temp(*this);
        for (int i = 0;i < _row;++i)
        {
            for (int j = 0;j < _column;++j)
            {
                r = get(i, j);
                ub = Ub->get(i, j);
                lb = Lb->get(i, j);
                if (ub < lb) {
                    throw std::invalid_argument("project impossible to have a value for the projection, ub>lb");
                }
                r = (ub - r) * (r > ub) + (lb - r) * (r < lb) + r; // permet de ne pas faire de branchement if.
                temp.set(i, j, r);
            }
        }
        this->set(&temp);
    }
    else {
        throw std::invalid_argument("project Matrix not at the same place");
    }
    
}
void MatrixGPU::projectNeg()
{
    if (_GPU)
    {
        projectGPUNeg << <_numBlocks, _blockSize >> > (_matrixGPU, _N);
    }
    else if (!_GPU)
    {
        for (int i = 0; i < _row; ++i)
        {
            for (int j = 0; j < _column; ++j)
            {
                float r = get(i, j);
                r = (r > 0) * r;
                set(i, j, r);
            }
        }
    }

}
void MatrixGPU::projectPos()
{
   
    if (_GPU)
    {
        projectGPUPos <<<_numBlocks, _blockSize >> > (_matrixGPU, _N);
    }
    else if (!_GPU)
    {
        for (int i = 0; i < _row; ++i)
        {
            for (int j = 0; j < _column; ++j)
            {
                float r = get(i, j);
                r = (r > 0) * r;
                set(i, j, r);
            }
        }
    }
}



float MatrixGPU::sum() const
{
    if (_row == 0 || _column == 0) {
        return 0;
        //throw std::out_of_range("Empty Matrix");
    }
    if (_GPU) 
    {
        int numBlocks = _numBlocks;
        unsigned int n = _N;
        float odata = 0;
        float* d_odata;
        if (preallocation) {
            d_odata = _preallocation;
        }
        else {
            hipMalloc((void**)&d_odata, sizeof(float) * numBlocks);
        }


        switch (_blockSize) {
        case 512:
            SumMultiBlock<512> <<<numBlocks, _blockSize >> > (_matrixGPU, d_odata, n);
            sumMonoBlock<512> <<< 1, _blockSize >> > (d_odata, d_odata, numBlocks);
            break;
        case 256:
            SumMultiBlock<256> << <numBlocks, _blockSize >> > (_matrixGPU, d_odata, n);
            sumMonoBlock<256> << < 1, _blockSize >> > (d_odata, d_odata, numBlocks);
            break;
        case 128:
            SumMultiBlock<128> << <numBlocks, _blockSize >> > (_matrixGPU, d_odata, n);
            sumMonoBlock<128> << < 1, _blockSize >> > (d_odata, d_odata, numBlocks);
            break;
        case 64:
            SumMultiBlock< 64> << <numBlocks, _blockSize >> > (_matrixGPU, d_odata, n);
            sumMonoBlock< 64> << < 1, _blockSize >> > (d_odata, d_odata, numBlocks);
            break;
        case 32:
            SumMultiBlock< 32> << <numBlocks, _blockSize >> > (_matrixGPU, d_odata, n);
            sumMonoBlock< 32> << < 1, _blockSize >> > (d_odata, d_odata, numBlocks);
            break;
        case 16:
            SumMultiBlock< 16> << <numBlocks, _blockSize >> > (_matrixGPU, d_odata, n);
            sumMonoBlock< 16> << < 1, _blockSize >> > (d_odata, d_odata, numBlocks);
            break;
        case  8:
            SumMultiBlock<  8> << <numBlocks, _blockSize >> > (_matrixGPU, d_odata, n);
            sumMonoBlock< 8> << < 1, _blockSize >> > (d_odata, d_odata, numBlocks);
            break;
        case  4:
            SumMultiBlock<  4> << <numBlocks, _blockSize >> > (_matrixGPU, d_odata, n);
            sumMonoBlock<  4> << < 1, _blockSize >> > (d_odata, d_odata, numBlocks);
            break;
        case  2:
            SumMultiBlock<  2> << <numBlocks, _blockSize >> > (_matrixGPU, d_odata, n);
            sumMonoBlock<  2> << < 1, _blockSize >> > (d_odata, d_odata, numBlocks);
            break;
        case  1:
            SumMultiBlock<  1> << <numBlocks, _blockSize >> > (_matrixGPU, d_odata, n);
            sumMonoBlock<  1> << < 1, _blockSize >> > (d_odata, d_odata, numBlocks);
            break;
        }
        //hipDeviceSynchronize();
        hipMemcpy(&odata, d_odata, sizeof(float), hipMemcpyDeviceToHost);
        if (!preallocation) {
            hipFree(d_odata);
        }
        //std::cout << "sum " << odata << " " <<_blockSize << " " << numBlocks << std::endl;
        return odata;
    }
    else if (!_GPU)
    {
        float d = 0;
        float r = 0;
        for (int i = 0;i < _row;++i)
        {
            for (int j = 0;j < _column;++j)
            {
                r = get(i, j);
                d = d + r;
            }
        }
        return d;
    }
}

float MatrixGPU::sum(int begin, int end)
{
    if (begin < 0 || end < 0) {
        throw std::invalid_argument("sum indice must be positve");
    }
    if (begin > end ) {
        throw std::invalid_argument("sum begin must be smaller than end");
    }
    if (begin > _N || end > _N) {
        throw std::out_of_range("sum indice must smaller than N");
    }
    if (_row == 0 || _column == 0) {
        return 0;
        //throw std::out_of_range("Empty Matrix");
    }

    if (_GPU)
    {
        int numBlocks = _numBlocks;
        float odata = 0;
        float* d_odata;
        if (preallocation) {
            d_odata = _preallocation;
        }
        else {
            hipMalloc((void**)&d_odata, sizeof(float) * numBlocks);
        }


        switch (_blockSize) {
        case 512:
            SumMultiBlock<512> << <numBlocks, _blockSize >> > (_matrixGPU, d_odata, begin, end);
            sumMonoBlock<512> << < 1, _blockSize >> > (d_odata, d_odata, numBlocks);
            break;
        case 256:
            SumMultiBlock<256> << <numBlocks, _blockSize >> > (_matrixGPU, d_odata, begin, end);
            sumMonoBlock<256> << < 1, _blockSize >> > (d_odata, d_odata, numBlocks);
            break;
        case 128:
            SumMultiBlock<128> << <numBlocks, _blockSize >> > (_matrixGPU, d_odata, begin, end);
            sumMonoBlock<128> << < 1, _blockSize >> > (d_odata, d_odata, numBlocks);
            break;
        case 64:
            SumMultiBlock< 64> << <numBlocks, _blockSize >> > (_matrixGPU, d_odata, begin, end);
            sumMonoBlock< 64> << < 1, _blockSize >> > (d_odata, d_odata, numBlocks);
            break;
        case 32:
            SumMultiBlock< 32> << <numBlocks, _blockSize >> > (_matrixGPU, d_odata, begin, end);
            sumMonoBlock< 32> << < 1, _blockSize >> > (d_odata, d_odata, numBlocks);
            break;
        case 16:
            SumMultiBlock< 16> << <numBlocks, _blockSize >> > (_matrixGPU, d_odata, begin, end);
            sumMonoBlock< 16> << < 1, _blockSize >> > (d_odata, d_odata, numBlocks);
            break;
        case  8:
            SumMultiBlock<  8> << <numBlocks, _blockSize >> > (_matrixGPU, d_odata, begin, end);
            sumMonoBlock< 8> << < 1, _blockSize >> > (d_odata, d_odata, numBlocks);
            break;
        case  4:
            SumMultiBlock<  4> << <numBlocks, _blockSize >> > (_matrixGPU, d_odata, begin, end);
            sumMonoBlock<  4> << < 1, _blockSize >> > (d_odata, d_odata, numBlocks);
            break;
        case  2:
            SumMultiBlock<  2> << <numBlocks, _blockSize >> > (_matrixGPU, d_odata, begin, end);
            sumMonoBlock<  2> << < 1, _blockSize >> > (d_odata, d_odata, numBlocks);
            break;
        case  1:
            SumMultiBlock<  1> << <numBlocks, _blockSize >> > (_matrixGPU, d_odata, begin, end);
            sumMonoBlock<  1> << < 1, _blockSize >> > (d_odata, d_odata, numBlocks);
            break;
        }
        //hipDeviceSynchronize();
        hipMemcpy(&odata, d_odata, sizeof(float), hipMemcpyDeviceToHost);
        if (!preallocation) {
            hipFree(d_odata);
        }
        //std::cout << "sum " << odata << " " <<_blockSize << " " << numBlocks << std::endl;
        return odata;
    }
    else if (!_GPU)
    {
        float d = 0;
        float r = 0;
        for (int elem = begin; elem < end; ++elem)
        {
                r = _matrixCPU[elem];
                d = d + r;
        }
        return d;
    }
}

void MatrixGPU::sum(MatrixGPU* m)
{
    float s = 0;
     // on travaille sur les lignes 
    if ((_column != 1) || (_row != m->getNLin())) {
        throw std::invalid_argument("sum wrong dimension of the column vector ");
    }
    int col = m->getNCol();
    if (_GPU && m->getPos())
    {
        int numBlocks = _row;
        switch (_blockSize) {
        case 512:
            SumEachRow<512> << <numBlocks, _blockSize >> > (m->_matrixGPU, _matrixGPU, col);
            break;
        case 256:
            SumEachRow<256> << <numBlocks, _blockSize >> > (m->_matrixGPU, _matrixGPU, col);
            break;
        case 128:
            SumEachRow<128> << <numBlocks, _blockSize >> > (m->_matrixGPU, _matrixGPU, col);
            break;
        case 64:
            SumEachRow< 64> << <numBlocks, _blockSize >> > (m->_matrixGPU, _matrixGPU, col);
            break;
        case 32:
            SumEachRow< 32> << <numBlocks, _blockSize >> > (m->_matrixGPU, _matrixGPU, col);
            break;
        case 16:
            SumEachRow< 16> << <numBlocks, _blockSize >> > (m->_matrixGPU, _matrixGPU, col);
            break;
        case  8:
            SumEachRow<  8> << <numBlocks, _blockSize >> > (m->_matrixGPU, _matrixGPU, col);
            break;
        case  4:
            SumEachRow<  4> << <numBlocks, _blockSize >> > (m->_matrixGPU, _matrixGPU, col);
            break;
        case  2:
            SumEachRow<  2> << <numBlocks, _blockSize >> > (m->_matrixGPU, _matrixGPU, col);
            break;
        case  1:
            SumEachRow<  1> << <numBlocks, _blockSize >> > (m->_matrixGPU, _matrixGPU, col);
            break;
        }
    }
    else if (!_GPU && !(m->getPos()))
    {
        for (int i = 0; i < _row;i++)
        {
            s = 0;
            for (int j = 0; j < col;j++)
            {
                s = s + m->get(i, j);
            }
            set(i, 0, s);
        }
    }
    else {
        throw std::invalid_argument("sum Matrix not at the same place");
    }
}

float MatrixGPU::distance2() {

    if (_GPU ) //&& m->getPos())
    {
        int numBlocks = _numBlocks;
        unsigned int n = _N;
        float* d_odata;
        if (preallocation) {
            d_odata = _preallocation;
        }
        else {
            hipMalloc((void**)&d_odata, sizeof(float) * numBlocks);
        }
        float odata = 0;
        
        
        //std::cout << _numBlocks << std::endl;
 
        switch (_blockSize) {
        case 512:
            distanceMultiBlock<512> << <numBlocks, _blockSize >> > (_matrixGPU, d_odata, n);
            sumMonoBlock<512> << < 1, _blockSize >> > (d_odata, d_odata, numBlocks);
        break;
        case 256:
            distanceMultiBlock<256> <<<numBlocks, _blockSize >> > (_matrixGPU, d_odata, n);
            sumMonoBlock<256> <<< 1, _blockSize >> > (d_odata, d_odata, numBlocks);
        break;
        case 128:
            distanceMultiBlock<128> <<<numBlocks, _blockSize >> > (_matrixGPU, d_odata, n);
            sumMonoBlock<128> <<< 1, _blockSize >> > (d_odata, d_odata, numBlocks);
        break;
        case 64:
            distanceMultiBlock< 64> <<<numBlocks, _blockSize >> > (_matrixGPU, d_odata, n);
            sumMonoBlock< 64> <<< 1, _blockSize >> > (d_odata, d_odata, numBlocks);
            break;
        case 32:
            distanceMultiBlock< 32> <<<numBlocks, _blockSize >> > (_matrixGPU, d_odata, n);
            sumMonoBlock< 32> <<< 1, _blockSize >> > (d_odata, d_odata, numBlocks);
            break;
        case 16:
            distanceMultiBlock< 16> <<<numBlocks, _blockSize >> > (_matrixGPU, d_odata, n);
            sumMonoBlock< 16> <<< 1, _blockSize >> > (d_odata, d_odata, numBlocks);
            break;
        case  8:
            distanceMultiBlock<  8> <<<numBlocks, _blockSize >> > (_matrixGPU, d_odata, n);
            sumMonoBlock< 8> <<< 1, _blockSize >> > (d_odata, d_odata, numBlocks);
            break;
        case  4:
            distanceMultiBlock<  4> <<<numBlocks, _blockSize >> > (_matrixGPU, d_odata, n);
            sumMonoBlock<  4> <<< 1, _blockSize >> > (d_odata, d_odata, numBlocks);
            break;
        case  2:
            distanceMultiBlock<  2> <<<numBlocks, _blockSize >> > (_matrixGPU, d_odata, n);
            sumMonoBlock<  2> <<< 1, _blockSize >> > (d_odata, d_odata, numBlocks);
            break;
        case  1:
            distanceMultiBlock<  1> <<<numBlocks, _blockSize >>> (_matrixGPU, d_odata, n);
            sumMonoBlock<  1> <<< 1, _blockSize >> > (d_odata, d_odata, numBlocks);
            break; 
        }
        //hipDeviceSynchronize();
        hipMemcpy(&odata, d_odata, sizeof(float), hipMemcpyDeviceToHost);
        if (!preallocation) {
            hipFree(d_odata);
        }
        return sqrtf(odata);
    }
    else if (!_GPU)// && !(m->getPos()))
    {
        float d = 0;
        float r = 0;
        for (int i = 0;i < _row;++i)
        {
            for (int j = 0;j < _column;++j)
            {
                r = get(i, j);// -m->get(i, j);
                d = d + r * r;
            }
        }
        return sqrtf(d); 
    }
}



///////////////////////////////////////////////////////////////////////////////
// Display MatrixGPU contents
///////////////////////////////////////////////////////////////////////////////
void MatrixGPU::display(bool force) 
{   
    bool transfert = false;
    if (this) {
        if (_GPU && !force ) {
           std::cout << " Matrix stockee sur GPU, faire le transfertCPU avant d'afficher " << std::endl;
        }
        else {
            if (_row == 0 || _column == 0)
            {
                std::cout << "matrix vide " << std::endl;
            }
            if (_GPU) {
                transferCPU();
                transfert = true;
            }
            if (_column == 1) {
                std::cout << " transpose  : ";
                for (int i = 0;i < _row;++i)
                {
                    for (int j = 0;j < _column;++j)
                    {
                        float value = get(i, j);
                        std::cout << std::setprecision(7) << value;
                        //std::cout << std::fixed << std::setprecision(2) << value;
                        std::cout << " ";
                    }
                }
                std::cout << std::endl;
            }
            else {
                for (int i = 0;i < _row;++i)
                {
                    for (int j = 0;j < _column;++j)
                    {
                        float value = get(i, j);
                        std::cout << std::setprecision(7) << value;
                        //std::cout << std::fixed << std::setprecision(3) << value;
                        if (j != _column - 1) std::cout << " ";
                    }

                    std::cout << std::endl;
                }
                std::cout << std::endl;
            }
            if (transfert) {
                transferGPU();
            }
        }
    }
    else 
    {
        std::cout << "matrix non definie " << std::endl;
    }
}

void MatrixGPU::displayBloc(int iBegin, int iEnd, int jBegin, int jEnd, bool force)
{
    if ((iBegin < 0) || (jBegin < 0) || iEnd > _row || jEnd > _column) {
        throw std::out_of_range("displayBloc index out of bounds");
    } if ((iBegin > iEnd) || (jBegin > jEnd)) {
        throw std::invalid_argument("displayBloc xBegin must be smaller than xEnd");
    }
    bool transfert = false;
    if (this) {
        if (_GPU && !force) {
            std::cout << " Matrix stockee sur GPU, faire le transfertCPU avant d'afficher " << std::endl;
        }
        else {
            if (_row == 0 || _column == 0)
            {
                std::cout << "matrix vide " << std::endl;
            }
            if (_GPU) {
                transferCPU();
                transfert = true;
            }
            if (jEnd - jBegin == 1 ) {
                std::cout << " transpose  : ";
                for (int i = iBegin; i < iEnd; ++i)
                {
                    float value = get(i, jBegin);
                    std::cout << std::setprecision(7) << value;
                    std::cout << " ";
                }
                std::cout << std::endl;
            }
            else {
                for (int i = iBegin; i < iEnd; ++i)
                {
                    for (int j = jBegin; j < jEnd; ++j)
                    {
                        float value = get(i, j);
                        std::cout << std::setprecision(7) << value;
                        if (j != jEnd - 1) {
                            std::cout << " ";
                        }
                    }

                    std::cout << std::endl;
                }
                std::cout << std::endl;
            }
            if (transfert) {
                transferGPU();
            }
        }
    }
    else
    {
        std::cout << "matrix non definie " << std::endl;
    }
}

void MatrixGPU::swapLine(int line1, int line2)
{
    if (_GPU) {
        swapLineGJ << <_numBlocks, _blockSize >> > (_matrixGPU, line1, line2, _column);// swap des lignes
    }
    else {
        float temp = 0;
        for (int i = 0; i < _column; i++) {
            temp = get(line1, i);
            set(line1, i, get(line2, i));
            set(line2, i, temp);
        }
    }
}




///////////////////////////////////////////////////////////////////////////////
// Destructor
///////////////////////////////////////////////////////////////////////////////
MatrixGPU::~MatrixGPU()
{
    #ifdef DEBUG_DESTRUCTOR
        std::cout << "destruction matrix " << _matrixGPU << std::endl;
    #endif // DEBUG_DESTRUCTOR
    if (_preallocationFloat != nullptr) {
        hipHostFree(_preallocationFloat);
        _preallocationFloat = nullptr;
    }
    if (_preallocation != nullptr) {
        hipFree(_preallocation);
        _preallocation = nullptr;
    }
    if (_matrixGPU != nullptr) {
        hipFree(_matrixGPU);
        _matrixGPU = nullptr;
    }
     DELETEA(_matrixCPU);
    
}



void MatrixGPU::saveCSV(const std::string& filename, std::ios_base::openmode mode, int trans) const
{
    if (_GPU) {
        throw std::domain_error("saveCSV : Matrix on GPU");
    }
    std::ofstream myfile;
    myfile.open(filename, mode);
    myfile.precision(50);
    if (!trans) {
        for (int i = 0; i < _row; i++) {
            for (int j = 0; j < _column;j++) {
                myfile << get(i, j) << ";";
            }
            myfile << "\n";
        }
    }
    else {
        for (int j = 0; j < _column;j++) {
            for (int i = 0; i < _row; i++) {
                myfile << get(i, j) << ";";
            }
            myfile << "\n";
        }
    }

    myfile.close();
}

void MatrixGPU::saveCSVForce(const std::string& filename, std::ios_base::openmode mode, int trans)
{
    int transfert = 0;
    if (_GPU) {
        transfert = 1;
        transferCPU();
    }
    std::ofstream myfile;
    myfile.open(filename, mode);
    myfile.precision(10);
    if (!trans) {
        for (int i = 0; i < _row; i++) {
            for (int j = 0; j < _column; j++) {
                myfile << get(i, j) << ";";
            }
            myfile << "\n";
        }
    }
    else {
        for (int j = 0; j < _column; j++) {
            for (int i = 0; i < _row; i++) {
                myfile << get(i, j) << ";";
            }
            myfile << "\n";
        }
    }

    myfile.close();
    if (transfert) {
        transferGPU();
    }
}


///////////////////////////////////////////////////////////////////////////////
// Fonction globale
///////////////////////////////////////////////////////////////////////////////


__global__ void setup_kernel(hiprandState* state) {

    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    hiprand_init(1234, idx, 0, &state[idx]);
}


__global__ void generate_kernel(hiprandState* my_curandstate, float* result, float eps, const unsigned int N) {

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int step = blockDim.x * gridDim.x;
    for (int i = index; i < N; i += step)
    {
        result[i] = (2*hiprand_uniform(my_curandstate + i)-1) * eps;
    }
}





__global__ void setGPU(float* mat1, float* mat2, int N) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int step = blockDim.x * gridDim.x;
    for (int i = index; i < N; i += step)
    {
        mat1[i] = mat2[i];
    }
}
__global__ void setGPUFD(float* mat1, double* mat2, int N) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int step = blockDim.x * gridDim.x;
    for (int i = index; i < N; i += step)
    {
        mat1[i] = mat2[i];
    }
}
__global__ void setGPUDF(double* mat1, float* mat2, int N) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int step = blockDim.x * gridDim.x;
    for (int i = index; i < N; i += step)
    {
        mat1[i] = (double) mat2[i];
    }
}


__global__ void setGPU(float* mat1, const float value, int N) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int step = blockDim.x * gridDim.x;
    for (int i = index; i < N; i += step)
    {
        mat1[i] = value;
    }
}
__global__ void setGPUunique(float* mat1, const float value, int pos) {
    int index = threadIdx.x;
    if (index == 0) {
        mat1[pos] = value;
    }

}

__global__ void setTransGPU(float* mat1, float* matToTrans, const int column, const int row) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int step = blockDim.x * gridDim.x;
    const int N = column * row;

    for (int e = index; e < N; e += step)
    {
        int i = e / column;
        int j = e % column;
        mat1[e] = matToTrans[j * row + i];

    }
}

__global__ void setColGPU(float* mat1, float* mat2, const int numCol, const int column, const int row, const int offset) {
    
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int step = blockDim.x * gridDim.x;
    for (int i = index; i < row; i += step)
    {
        mat1[i] = i < offset ? 0 : mat2[i*column + numCol];
    }

}

__global__ void setEyesGPU(float* mat2, const float value, const int col, const int row) 
{
    int N = row * col;
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int step = blockDim.x * gridDim.x;

    for (int l = index; l < N; l+=step) {
        int i = l / col;
        int j = l % col;
        mat2[l] = (i==j) ? value : 0; // pas coalescent, mais bon...
    }
}
__global__ void setEyesGPU(float* mat2, float* mat1, const int col, const int row)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int step = blockDim.x * gridDim.x;
    int N = row * col;
    for (int l = index; l < N; l += step) {
        int i = l / col;
        int j = l % col;
        mat2[l] = (i == j) ? mat1[i] : 0; // pas coalescent, mais bon...
    }
}


__global__ void SetBlocGPU(float* out, float* in, int ibegin, int iend, int jbegin, int jend, int col)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int step = blockDim.x * gridDim.x;
    int offset = jbegin + ibegin * col;
    int N = (jend - jbegin) * (iend - ibegin);

    for (int j = index; j < N; j += step)
    {
        int rowLoc = j / (jend - jbegin);
        int colLoc = j % (jend - jbegin);
        int GlobalInd = offset + rowLoc *col + colLoc;
        out[GlobalInd] = in[j];
    }
}

__global__ void SetBlocGPU(float* out, float value, int ibegin, int iend, int jbegin, int jend, int col)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int step = blockDim.x * gridDim.x;
    int offset = jbegin + ibegin * col;
    int N = (jend - jbegin) * (iend - ibegin);

    for (int j = index; j < N; j += step)
    {
        int rowLoc = j / (jend - jbegin);
        int colLoc = j % (jend - jbegin);
        int GlobalInd = offset + rowLoc * col + colLoc;
        out[GlobalInd] = value;
    }
}

__global__ void SetBlocGPU(float* out, float* in, int ibegin, int iend, int jbegin, int jend, int col, float factor)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int step = blockDim.x * gridDim.x;
    int offset = jbegin + ibegin * col;
    int N = (jend - jbegin) * (iend - ibegin);

    for (int j = index; j < N; j += step)
    {
        int rowLoc = j / (jend - jbegin);
        int colLoc = j % (jend - jbegin);
        int GlobalInd = offset + rowLoc * col + colLoc;
        out[GlobalInd] = factor * in[j];
    }
}

/*__global__ void SetBlocGPU(float* out, float* in, int ibegin, int iend, int jbegin, int jend, int col, float factor) // fait que la première ligne
{
    int indexX = threadIdx.x + blockIdx.x * blockDim.x;
    int stepX = blockDim.x * gridDim.x;
    int indexY = threadIdx.y + blockIdx.y * blockDim.y;
    int stepY = blockDim.y * gridDim.y;

    for (int j = indexX + jbegin; j < jend; j += stepX)
    {
        for (int i = indexY + ibegin; i < iend; i += stepY)
        {
            out[j + i * col] = factor * in[indexX + indexY * col];
        }
    }
}*/




__global__ void replaceGPU(float* mat,const float previous, const float newValue,const int N) 
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int step = blockDim.x * gridDim.x;
    for (int i = index; i < N; i += step)
    {
        mat[i] = (mat[i] == previous) * (newValue-mat[i]) + mat[i];
    }
}




__global__ void addGPU(float* mat, float c, int N) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int step = blockDim.x * gridDim.x;
    for (int i = index; i < N; i += step)
    {
        mat[i] = mat[i] + c;
    }
}
__global__ void addGPU(float* mat1, float* mat2, float c, int N) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int step = blockDim.x * gridDim.x;
    for (int i = index; i < N; i += step)
    {
        mat1[i] = mat2[i] + c;
    }
}
__global__ void addGPU(float* mat1, float* mat2, int N) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int step = blockDim.x * gridDim.x;
    for (int i = index; i < N; i += step)
    {
        mat1[i] = mat1[i] + mat2[i];
    }
}
__global__ void addGPU(float* mat1, float* mat2, float* mat3, int N) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int step = blockDim.x * gridDim.x;
    for (int i = index; i < N; i += step)
    {
        mat1[i] = mat2[i] + mat3[i];
    }
}

__global__ void addVectorGPU1(float* mat1, float* vect, const int n, int N) //vecteur colonne
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int step = blockDim.x * gridDim.x;
    for (int i = index; i < N; i += step)
    {
        int k = i / n; // division entière
        mat1[i] = mat1[i] + vect[k];
    }

}
__global__ void addVectorGPU2(float* mat1, float* vect, const int n, int N) // vecteur ligne
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int step = blockDim.x * gridDim.x;

    for (int i = index; i < N; i += step)
    {
        int k = i % n; // modulo
        mat1[i] = mat1[i] + vect[k];
    }


}

__global__ void addTransGPU(float* out, float* mat1, float* mat2, const int col, const int line, int N) 
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int step = blockDim.x * gridDim.x;
    for (int l = index; l < N; l += step)
    {
        int i = l / col;
        int j = l % col;
        int k = i + j * line;
        out[l] = mat1[l] + mat2[k];
    }
}

__global__ void substractGPU(float* mat1, float* mat2, int N) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int step = blockDim.x * gridDim.x;

    for (int i = index; i < N; i += step)
    {
        mat1[i] = mat1[i] - mat2[i];
    }
}
__global__ void substractGPU(float* mat1, float* mat2, float* mat3, int N) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int step = blockDim.x * gridDim.x;
    for (int i = index; i < N; i += step)
    {
        mat1[i] = mat2[i] - mat3[i];
    }
}

__global__ void substractVectorGPU1(float* mat1, float* vect, const int n, int N) //vecteur colonne
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int step = blockDim.x * gridDim.x;
    for (int i = index; i < N; i += step)
    {
        int k = i / n; // division entière
        mat1[i] = mat1[i] - vect[k];
    }

}
__global__ void substractVectorGPU2(float* mat1, float* vect, const int n, int N) // vecteur ligne
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int step = blockDim.x * gridDim.x;
    for (int i = index; i < N; i += step)
    {
        int k = i % n; // modulo
        mat1[i] = mat1[i] - vect[k];
    }

}

__global__ void substractTransGPU(float* out, float* mat1, float* mat2, const int col, const int line, int N)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int step = blockDim.x * gridDim.x;
    for (int l = index; l < N; l += step)
    {
        int i = l / col;
        int j = l % col;
        int k = i + j * line;
        out[l] = mat1[l] - mat2[k];
    }
}

__global__ void multiplyGPU(float* mat, const float c, int N) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int step = blockDim.x * gridDim.x;
    for (int i = index; i < N; i += step)
    {
        mat[i] = mat[i] * c;
    }
}

__global__ void multiplyTGPU(float* mat1, float* mat2, int N) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int step = blockDim.x * gridDim.x;
    for (int i = index; i < N; i += step)
    {
        mat1[i] = mat1[i] * mat2[i];
    }
}
__global__ void multiplyTGPU(float* mat1, float* mat2, float* mat3, int N) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int step = blockDim.x * gridDim.x;
    for (int i = index; i < N; i += step)
    {
        mat1[i] = mat2[i] * mat3[i];
    }
}

__global__ void divideGPU(float* mat, const float c, int N) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int step = blockDim.x * gridDim.x;
    for (int i = index; i < N; i += step)
    {
        mat[i] = mat[i] / c;
    }
}
__global__ void divideGPU(float* mat1, float* mat2, int N) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int step = blockDim.x * gridDim.x;
    for (int i = index; i < N; i += step)
    {
        mat1[i] = mat1[i] / mat2[i];
    }
}

__global__ void moyGPU1(float* res, float* mat1, float* nb, const int line, const int column) //vecteur ligne
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int step = blockDim.x * gridDim.x;
     
    for (int i = index; i < column; i += step)
    {
        float s = 0.0;
        for (int j = 0; j < line; j++)
        {
            s += mat1[i + column *j];
        } 
        res[i] = s / nb[i];
    }

}
__global__ void moyGPU2(float* res, float* mat1, float* nb, const int line, const int column) // vecteur colonne
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int step = blockDim.x * gridDim.x;
    
    for (int i = index; i < line; i += step)
    {
        float s = 0.0;
        for (int j = 0; j < column; j++)
        {
            s +=  mat1[i*column + j];
        }
        res[i] = s /nb[i];
    }
}

__global__ void projectGPU(float* mat, float* Lb, float* Ub, int N) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int step = blockDim.x * gridDim.x;
    for (int i = index; i < N; i += step)
    {
        float r = mat[i];
        float ub = Ub[i];
        float lb = Lb[i];
        r = (ub - r) * (r > ub) + (lb - r) * (r < lb) + r;
        mat[i] = r;//(Ub[i] - mat[i])* (mat[i] > Ub[i]) + (Lb[i] - mat[i]) * (mat[i] < Lb[i]) + mat[i];
    }
}

__global__ void projectGPUPos(float* mat, int N) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int step = blockDim.x * gridDim.x;
    for (int i = index; i < N; i += step)
    {
        float r = mat[i];
        mat[i] = (r > 0) * r;
    }
}

__global__ void projectGPUNeg(float* mat, int N) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int step = blockDim.x * gridDim.x;
    for (int i = index; i < N; i += step)
    {
        float r = mat[i];
        mat[i] = (r < 0) * r;
    }
}


__global__ void sumGPU(float* res, float* mat1, const int line, const int column) //vecteur colonne
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int step = blockDim.x * gridDim.x;

    for (int i = index; i < line; i += step)
    {
        float s = 0.0;
        for (int j = 0; j < column; j++)
        {
            s += mat1[i*column + j];
        }
        res[i] = s;
    }
}

__global__ void sumGPU2(float* res, float* mat1, const int line) //vecteur ligne
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if(index==0)
    {
        float s = 0.0;
        for (int j = 0; j < line; j++)
        {
            s += mat1[j];
        }
        
        *res = s ;
    }
}



__device__ int sumCommSingleWarp(volatile float* shArr) {
    int idx = threadIdx.x % warpSize; //the lane index in the warp
    if (idx < 16) {
        shArr[idx] += shArr[idx + 16];
        shArr[idx] += shArr[idx + 8];
        shArr[idx] += shArr[idx + 4];
        shArr[idx] += shArr[idx + 2];
        shArr[idx] += shArr[idx + 1];
    }
    return shArr[0];
}

template <unsigned int blockSize>
__global__ void sumMonoBlock(float* g_idata, float* g_odata, unsigned int n) {
    
    int thIdx = threadIdx.x;
    float sum = 0;
    for (int i = thIdx; i < n; i += blockSize)
        sum += g_idata[i];
    __shared__ float r[blockSize];
    r[thIdx] = sum;
    __syncthreads();
    if (blockSize >= 512) {
        if (thIdx < 256) {
            r[thIdx] += r[thIdx + 256];
        }
        __syncthreads();
    }
    if (blockSize >= 256) {
        if (thIdx < 128) {
            r[thIdx] += r[thIdx + 128];
        }
        __syncthreads();
    }
    if (blockSize >= 128) {
        if (thIdx < 64) {
            r[thIdx] += r[thIdx + 64];
        }
        __syncthreads();
    }
    //if (blockSize >= 64) {
    //    if (thIdx < 32) {
            warpReduce<blockSize>(r, thIdx);
    //    }
    // }
    //else if (blockSize >= 32) { // cas blockSize = 32
       // warpReduce<blockSize>(r, thIdx);
   // }
    __syncthreads;
    if (thIdx == 0) {
         *g_odata = r[0];
    }
       
    
}


template <unsigned int blockSize>
__global__ void SumMultiBlock(float* g_idata, float* g_odata, unsigned int n) {

    __shared__ float shArr[blockSize];
    int thIdx = threadIdx.x;

    int gthIdx = thIdx + blockIdx.x * blockSize;
    const int gridSize = blockSize * gridDim.x;
    float sum = 0;
    for (int i = gthIdx; i < n; i += gridSize)
       sum += g_idata[i];
    
    shArr[thIdx] = sum;
    __syncthreads();
    for (int size = blockSize / 2; size > 0; size /= 2) { //uniform
        if (thIdx < size)
            shArr[thIdx] += shArr[thIdx + size];
        __syncthreads();
    }
    if (thIdx == 0) {
        g_odata[blockIdx.x] = shArr[0];
    }
}

template <unsigned int blockSize>
__global__ void SumMultiBlock(float* g_idata, float* g_odata, unsigned int begin, unsigned int end) {
    __shared__ float shArr[blockSize];
    int thIdx = threadIdx.x;
    int gthIdx = thIdx + blockIdx.x * blockSize;
    const int gridSize = blockSize * gridDim.x;
    float sum = 0;

    for (int i = gthIdx + begin; i < end; i += gridSize) {
        sum += g_idata[i];
    }
        

    shArr[thIdx] = sum;
    __syncthreads();
    for (int size = blockSize / 2; size > 0; size /= 2) { //uniform
        if (thIdx < size)
            shArr[thIdx] += shArr[thIdx + size];
        __syncthreads();
    }
    if (thIdx == 0) {
        g_odata[blockIdx.x] = shArr[0];
    }
}



template <unsigned int blockSize>
__global__ void SumEachRow(float* g_idata, float* g_odata, const int nCol) {
    __shared__ float shArr[blockSize];
    int thIdx = threadIdx.x;
    int row = blockIdx.x;
    int idBegin = thIdx + row * nCol;
    int idEnd = (row + 1) * nCol;
    int step = blockDim.x;

    float sum = 0;
    for (int i = idBegin; i < idEnd; i += step) {
        sum += g_idata[i]; 
    }
        

    shArr[thIdx] = sum;
    __syncthreads();
    for (int size = blockSize / 2; size > 0; size /= 2) { //uniform
        if (thIdx < size)
            shArr[thIdx] += shArr[thIdx + size];
        __syncthreads();
    }
    if (thIdx == 0) {
        g_odata[blockIdx.x] = shArr[0];
    }
}

template <unsigned int blockSize>
__global__ void distanceMultiBlock(float* g_idata, float* g_odata, unsigned int n) {
    int thIdx = threadIdx.x;
    int gthIdx = thIdx + blockIdx.x * blockSize;
    const int gridSize = blockSize * gridDim.x;
    float sum = 0;
    for (int i = gthIdx; i < n; i += gridSize)
        sum += (g_idata[i] * g_idata[i]);
    __shared__ float shArr[blockSize];
    shArr[thIdx] = sum;
    __syncthreads();
    for (int size = blockSize / 2; size > 0; size /= 2) { //uniform
        if (thIdx < size)
            shArr[thIdx] += shArr[thIdx + size];
        __syncthreads();
    }
    if (thIdx == 0)
        g_odata[blockIdx.x] = shArr[0];
   
}



template <unsigned int blockSize>
__global__ void distanceMultiBlock(float* g_idata, float* g_idata2, float* g_odata, unsigned int n) {
    int thIdx = threadIdx.x;
    int gthIdx = thIdx + blockIdx.x * blockSize;
    const int gridSize = blockSize * gridDim.x;
    float sum = 0;
    for (int i = gthIdx; i < n; i += gridSize)
        sum += ((g_idata[i]- g_idata2[i]) * (g_idata[i] - g_idata2[i]));
    __shared__ float shArr[blockSize];
    shArr[thIdx] = sum;
    __syncthreads();
    for (int size = blockSize / 2; size > 0; size /= 2) { //uniform
        if (thIdx < size)
            shArr[thIdx] += shArr[thIdx + size];
        __syncthreads();
    }
    if (thIdx == 0)
        g_odata[blockIdx.x] = shArr[0];

}


template <unsigned int blockSize>
__device__ void warpReduceMaxPos(volatile float* r, volatile int* pos){
    int idx = threadIdx.x % warpSize; //the lane index in the warp

    if (idx < 32 && blockSize >= 64) {
        pos[idx] = r[idx + 32] > r[idx] ? pos[idx + 32] : pos[idx];
        r[idx] = r[idx + 32] > r[idx] ? r[idx + 32] : r[idx];//r[idx + 16] > r[idx] ? r[idx + 16] : r[idx];//r[idx + 16] * (r[idx + 16] > r[idx]) + r[idx] * (r[idx] <= r[idx + 16]);
    }
    __syncwarp();
    if (idx < 16 && blockSize >= 32) {
        pos[idx] = r[idx + 16] > r[idx] ? pos[idx + 16] : pos[idx];
        r[idx] = r[idx + 16] > r[idx] ? r[idx + 16] : r[idx];//r[idx + 16] > r[idx] ? r[idx + 16] : r[idx];//r[idx + 16] * (r[idx + 16] > r[idx]) + r[idx] * (r[idx] <= r[idx + 16]);
    }
    __syncwarp();
    if (idx <  8 && blockSize >= 16) {
        pos[idx] = r[idx + 8] > r[idx] ? pos[idx + 8] : pos[idx];
        r[idx] = r[idx + 8] > r[idx] ? r[idx + 8] : r[idx];//r[idx +  8] > r[idx] ? r[idx +  8] : r[idx];//r[idx +  8] * (r[idx +  8] > r[idx]) + r[idx] * (r[idx] <= r[idx +  8]);
    }
    __syncwarp();
    if (idx < 4 && blockSize >=  8) {
        pos[idx] = r[idx + 4] > r[idx] ? pos[idx + 4] : pos[idx];
        r[idx] = r[idx + 4] > r[idx] ? r[idx + 4] : r[idx];//r[idx +  4] > r[idx] ? r[idx +  4] : r[idx];//r[idx +  4] * (r[idx +  4] > r[idx]) + r[idx] * (r[idx] <= r[idx +  4]);
    }
    __syncwarp();
    if (idx < 2 && blockSize >=  4) {
        pos[idx] = r[idx + 2] > r[idx] ? pos[idx + 2] : pos[idx];
        r[idx] = r[idx + 2] > r[idx] ? r[idx + 2] : r[idx];//r[idx +  2] > r[idx] ? r[idx +  2] : r[idx];//r[idx +  2] * (r[idx +  2] > r[idx]) + r[idx] * (r[idx] <= r[idx +  2]);
    }
    __syncwarp();
    if (idx < 1 && blockSize >=  2) {
        pos[idx] = r[idx + 1] > r[idx] ? pos[idx + 1] : pos[idx];
        r[idx] = r[idx + 1] > r[idx] ? r[idx + 1] : r[idx];//r[idx +  1] > r[idx] ? r[idx +  1] : r[idx];//r[idx +  1] * (r[idx +  1] > r[idx]) + r[idx] * (r[idx] <= r[idx +  1]);
    }
}


template <unsigned int blockSize>
__global__ void maxMonoBlock(float* g_idata, float* g_odata, unsigned int n) {
    int thIdx = threadIdx.x;
    float max = 0;

    for (int i = thIdx; i < n; i += blockSize) {
        float s = g_idata[i];
        max = s > max ? s : max;// s>max ? s:max;//s * (s > max) + max * (max <= s);
    }
    __shared__ float shArr[blockSize];
    shArr[thIdx] = max;
    __syncthreads();
    if (blockSize >= 512) {
        if (thIdx < 256) {
            shArr[thIdx] = shArr[thIdx + 256] > shArr[thIdx] ? shArr[thIdx + 256] : shArr[thIdx]; //shArr[thIdx + size] > shArr[thIdx] ? shArr[thIdx + size] : shArr[thIdx];//shArr[thIdx + size] * (shArr[thIdx + size] > shArr[thIdx]) + shArr[thIdx] * (shArr[thIdx] <= shArr[thIdx + size]); 

        }
        __syncthreads();
    }
    if (blockSize >= 256) {
        if (thIdx < 128) {
            shArr[thIdx] = shArr[thIdx + 128] > shArr[thIdx] ? shArr[thIdx + 128] : shArr[thIdx]; //shArr[thIdx + size] > shArr[thIdx] ? shArr[thIdx + size] : shArr[thIdx];//shArr[thIdx + size] * (shArr[thIdx + size] > shArr[thIdx]) + shArr[thIdx] * (shArr[thIdx] <= shArr[thIdx + size]); 

        }
        __syncthreads();
    }
    if (blockSize >= 128) {
        if (thIdx < 64) {
            shArr[thIdx] = shArr[thIdx + 64] > shArr[thIdx] ? shArr[thIdx + 64] : shArr[thIdx]; //shArr[thIdx + size] > shArr[thIdx] ? shArr[thIdx + size] : shArr[thIdx];//shArr[thIdx + size] * (shArr[thIdx + size] > shArr[thIdx]) + shArr[thIdx] * (shArr[thIdx] <= shArr[thIdx + size]); 

        }
        __syncthreads();
    }
    if (blockSize >= 64) {
        if (thIdx < 32) {
            warpReduceMax<blockSize>(shArr, thIdx);
        }
    }
    if (blockSize >= 32 && blockSize < 64) {
       warpReduceMax<blockSize>(shArr, thIdx);
    }
    __syncthreads;

    if (thIdx == 0)
        *g_odata = shArr[0];
    
}

template <unsigned int blockSize>
__global__ void maxMultiBlock(float* g_idata, float* g_odata, unsigned int n) {
    int thIdx = threadIdx.x;
    int gthIdx = thIdx + blockIdx.x * blockSize;
    const int gridSize = blockSize * gridDim.x;
    float max = 0;
    for (int i = gthIdx; i < n; i += gridSize) {
        float s = (g_idata[i] * g_idata[i]);
        max = s > max ? s : max;//s > max ? s : max; //s * (s > max) + max * (max <= s);
    }
    __shared__ float shArr[blockSize];
    shArr[thIdx] = max;
    __syncthreads();
    for (int size = blockSize / 2; size > 0; size /= 2) { //uniform
        if (thIdx < size)
            shArr[thIdx] = shArr[thIdx + size] > shArr[thIdx] ? shArr[thIdx + size] : shArr[thIdx]; //shArr[thIdx + size] > shArr[thIdx] ? shArr[thIdx + size] : shArr[thIdx];//shArr[thIdx + size] * (shArr[thIdx + size] > shArr[thIdx]) + shArr[thIdx] * (shArr[thIdx] <= shArr[thIdx + size]); 
        __syncthreads();
    }
    if (thIdx == 0)
        g_odata[blockIdx.x] = shArr[0];
}

template <unsigned int blockSize>
__global__ void maxMultiBlock(float* g_idata, float* g_odata, unsigned int n, int* pos) {
    int thIdx = threadIdx.x;
    int gthIdx = thIdx + blockIdx.x * blockSize;
    const int gridSize = blockSize * gridDim.x;
    float max = 0;
    int indice = 0;
    for (int i = gthIdx; i < n; i += gridSize) {
        float s = (g_idata[i] * g_idata[i]);
        indice = s > max ? i : indice;
        max = s > max ? s : max;//s > max ? s : max; //s * (s > max) + max * (max <= s);
    }
    __shared__ float shArr[blockSize];
    __shared__ float shPos[blockSize];
    shArr[thIdx] = max;
    shPos[thIdx] = indice;
    __syncthreads();
    for (int size = blockSize / 2; size > 0; size /= 2) { //can unroll the for loop
        if (thIdx < size) {
            shPos[thIdx] = shArr[thIdx + size] > shArr[thIdx] ? shPos[thIdx + size] : shPos[thIdx];
            shArr[thIdx] = shArr[thIdx + size] > shArr[thIdx] ? shArr[thIdx + size] : shArr[thIdx]; 
            //shArr[thIdx + size] > shArr[thIdx] ? shArr[thIdx + size] : shArr[thIdx];//shArr[thIdx + size] * (shArr[thIdx + size] > shArr[thIdx]) + shArr[thIdx] * (shArr[thIdx] <= shArr[thIdx + size]); 
        }
           
        __syncthreads();
    }
    if (thIdx == 0) {
        g_odata[blockIdx.x] = shArr[0];
        pos[blockIdx.x] = shPos[0];

    }
}
       

template <unsigned int blockSize>
__global__ void maxMonoBlock(float* g_idata, float* g_odata, unsigned int n, int* pos) {
    int thIdx = threadIdx.x;
    float max = 0;
    int indice = 0;
    for (int i = thIdx; i < n; i += blockSize) {
        float s = g_idata[i];
        indice = s > max ? pos[i] : indice;
        max = s > max ? s : max;// s>max ? s:max;//s * (s > max) + max * (max <= s);
    }
    __shared__ float shArr[blockSize];
    __shared__ int shPos[blockSize];
    shArr[thIdx] = max;
    shPos[thIdx] = indice;

    __syncthreads();
    if (blockSize >= 512) {
        if (thIdx < 256) {
            shPos[thIdx] = shArr[thIdx + 256] > shArr[thIdx] ? shPos[thIdx + 256] : shPos[thIdx];
            shArr[thIdx] = shArr[thIdx + 256] > shArr[thIdx] ? shArr[thIdx + 256] : shArr[thIdx]; //shArr[thIdx + size] > shArr[thIdx] ? shArr[thIdx + size] : shArr[thIdx];//shArr[thIdx + size] * (shArr[thIdx + size] > shArr[thIdx]) + shArr[thIdx] * (shArr[thIdx] <= shArr[thIdx + size]); 
        }
        __syncthreads();
    }
    if (blockSize >= 256) {
        if (thIdx < 128) {
            shPos[thIdx] = shArr[thIdx + 128] > shArr[thIdx] ? shPos[thIdx + 128] : shPos[thIdx];
            shArr[thIdx] = shArr[thIdx + 128] > shArr[thIdx] ? shArr[thIdx + 128] : shArr[thIdx]; //shArr[thIdx + size] > shArr[thIdx] ? shArr[thIdx + size] : shArr[thIdx];//shArr[thIdx + size] * (shArr[thIdx + size] > shArr[thIdx]) + shArr[thIdx] * (shArr[thIdx] <= shArr[thIdx + size]); 

        }
        __syncthreads();
    }
    if (blockSize >= 128) {
        if (thIdx < 64) {
            shPos[thIdx] = shArr[thIdx + 64] > shArr[thIdx] ? shPos[thIdx + 64] : shPos[thIdx];
            shArr[thIdx] = shArr[thIdx + 64] > shArr[thIdx] ? shArr[thIdx + 64] : shArr[thIdx]; //shArr[thIdx + size] > shArr[thIdx] ? shArr[thIdx + size] : shArr[thIdx];//shArr[thIdx + size] * (shArr[thIdx + size] > shArr[thIdx]) + shArr[thIdx] * (shArr[thIdx] <= shArr[thIdx + size]); 
        }
        __syncthreads();
    }
    if (blockSize >= 64) {
        if (thIdx < 32) {
            warpReduceMaxPos<blockSize>(shArr, shPos);
        }
    }
    if (blockSize >= 32 && blockSize < 64) {
        warpReduceMaxPos<blockSize>(shArr, shPos);
    }
    __syncthreads();
   
    if (thIdx == 0) {
        *g_odata = shArr[0];
        *pos = shPos[0];
    }
       
}


template <unsigned int blockSize>
__global__ void maxMultiBlock(float* g_idata, float* g_idata2, float* g_odata, unsigned int n) {
    int thIdx = threadIdx.x;
    int gthIdx = thIdx + blockIdx.x * blockSize;
    const int gridSize = blockSize * gridDim.x;
    float max = 0;
    for (int i = gthIdx; i < n; i += gridSize) {
        float s = (g_idata[i] - g_idata2[i]);
        s = s * s;
        max = s > max ? s : max;//s > max ? s : max; //s * (s > max) + max * (max <= s);
    }
    __shared__ float shArr[blockSize];
    shArr[thIdx] = max;
    __syncthreads();
    for (int size = blockSize / 2; size > 0; size /= 2) { //uniform
        if (thIdx < size) {
            shArr[thIdx] = shArr[thIdx + size] > shArr[thIdx] ? shArr[thIdx + size] : shArr[thIdx]; //shArr[thIdx + size] > shArr[thIdx] ? shArr[thIdx + size] : shArr[thIdx];//shArr[thIdx + size] * (shArr[thIdx + size] > shArr[thIdx]) + shArr[thIdx] * (shArr[thIdx] <= shArr[thIdx + size]); 
        }
            
        __syncthreads();
    }
    if (thIdx == 0) {
        g_odata[blockIdx.x] = shArr[0];
    }
}


__global__ void normalisationGJ(float* mat, const int row, const int nCol, const float factor) 
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int step = blockDim.x * gridDim.x;
    for (int i = index; i < nCol; i += step)
    {
        mat[i + row * nCol] = mat[i + row * nCol] / factor;
    }


}

__global__ void swapLineGJ(float* mat, const int row1, const int row2, const int nCol) 
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int step = blockDim.x * gridDim.x;
    for (int i = index; i < nCol; i += step)
    {
        float temp = mat[i + row1 * nCol];
        float temp2 = mat[i + row2 * nCol]; 
        mat[i + row1 * nCol] = temp2; // or mat[i + row1 * nCol] = mat[i + row2 * nCol];
        mat[i + row2 * nCol] = temp;
    }
}

__global__ void eliminationGJ(float* mat, float* matAug, const int r, const int nRow, const int nCol) {

    // un bloc = une ligne, 
    int index = threadIdx.x;
    int row = blockIdx.x;
    int step = blockDim.x;
    __shared__ float shFactor;
    if (row != r) { // le bloc r ne fait rien... bah...
        if (index == 0) {
            shFactor = mat[row * nCol + r];
        }
        __syncthreads();
        for (int j = index; j < nCol; j+=step) {
            float value1 = mat[r * nCol + j];
            float oldvalue1 = mat[row * nCol + j];
            float oldvalue2 = matAug[row * nCol + j];
            float value2 = matAug[r * nCol + j];

            mat[row * nCol + j] = oldvalue1 - shFactor * value1;
            //matAug[row * nCol + j] -=  shFactor * value2;
            matAug[row * nCol + j] = oldvalue2 - shFactor * value2;
        }
    }
    

    /*int indexX = blockIdx.x * blockDim.x + threadIdx.x;
    int indexY = blockIdx.y * blockDim.y + threadIdx.y;
    int stepX = blockDim.x * gridDim.x;
    int stepY = blockDim.y * gridDim.y;


    for (int i = indexY; i < nRow; i += stepY)
    {
        if (i != r) {
            float factor = mat[i * nCol + r]; // ne doit pas changer tant que la ligne n'est pas fini
            for (int j = indexX; j < nCol; j += stepX)
            {
                if (j != r) {
                    float value1 = mat[r * nCol + j];
                    float value2 = matAug[r * nCol + j];

                    mat[i * nCol + j] = mat[i * nCol + j] - factor * value1;
                    matAug[i * nCol + j] = matAug[i * nCol + j] - factor * value2;
                }
            }
        }
    }
    __syncthreads();
    for (int i = indexY; i < nRow; i += stepY)
    {
        float factor = mat[i * nCol + r]; // ne doit pas changer tant que la ligne n'est pas fini
        if (i != r) {
            if (indexX == 1) {
                mat[i * nCol + r] = 0;
                matAug[i * nCol + r] = matAug[i * nCol + r] - factor * matAug[r * nCol + r];
            }
        }
    }*/
}



__global__ void initPermMatr(float* P, const int N) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int step = blockDim.x * gridDim.x;
    for (int i = index; i < (N+1); i += step)
    {
        P[i] = i*(i<N);
    }
}


__global__ void updatePermMatr(float* P, const int line1, const int line2, const int N) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int step = blockDim.x * gridDim.x;
    if (index == 0) {
        int inter = P[line1];
        P[line1] = P[line2];
        P[line2] = inter;
        P[N] = P[N] + 1;
    }
}


__global__ void updateLUPFactorization(float* A, const int col, const int N) {
    // un bloc par ligne i ?
    int index = threadIdx.x;
    int i = blockIdx.x;
    int step = blockDim.x;

    __shared__ float Aicol;

    if (i > col) { // les blocs trop petits ne font rien, en s'en fout ?
        if (index == 0) {
            Aicol = A[i * N + col] / A[col * N + col];
            A[i * N + col] = Aicol;
        }
        __syncthreads();
        for (int k = index + col + 1; k < N; k += step) {
            A[i * N + k] = A[i * N + k] - Aicol * A[col * N + k];
        }
    }
}





__global__ void setPermute(float* y, float* b, float* P, const int N) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int step = blockDim.x * gridDim.x;

    for (int i = index; i < N; i += step)
    {
        int indice = P[i];
        y[i] = b[indice]; // c'est absolument moche...
    }


}
__global__ void solveLowSys(float* A, float* y, const int iter, const int N) {
    int index = threadIdx.x;
    int step = blockDim.x;
    __shared__ float yiter;

    if (index == 0) {
        yiter = y[iter];
        
    }
    __syncthreads();
    for (int i = index + iter+1; i < N; i += step)
    {
        y[i] = y[i] - yiter * A[i * N + iter]; // moche ne faudrait-il pas stocker A^T ?

    }


}

__global__ void solveUpSys(float* A, float* y, const int iter, const int N) {
    int index = threadIdx.x;
    int step = blockDim.x;
    __shared__ float yiter;

    if (index == 0) {
        yiter = y[iter] / A[iter * N + iter];
        y[iter] = yiter;
    }
    __syncthreads();

    for (int i = index; i < iter; i += step)
    {
        y[i] = y[i] - yiter * A[i * N + iter]; // moche ne faudrait-il pas stocker A^T ?

    }


}


/* 
 int n = getNLin();
        setPermute << <_numBlocks, _blockSize >> > (_matrixGPU, b->_matrixGPU, P->_matrixGPU, n);
        for (int i = 0; i < n; i++) {
            solveLowSys <<< 1, _blockSize >>> (A->_matrixGPU, _matrixGPU, i, n);
        }


        for (int i = n - 1; i >= 0; i--)
        {
            solveUpSys << < 1, _blockSize >> > (A->_matrixGPU, _matrixGPU, i, n);
        }
*/

__global__ void solveSysGPU(float* A, float* y, const int N) {


    int index = threadIdx.x;
    int step = blockDim.x;
    extern __shared__ float ytemp[];


    for (int n = index; n < N; n += step)
    {
        ytemp[n] = y[n];
    }
    __syncthreads();
    for (int iter = 0; iter < N; iter++) {
        for (int i = index + iter + 1; i < N; i += step)
        {
            ytemp[i] = ytemp[i] - ytemp[iter] * A[i * N + iter]; // moche ne faudrait-il pas stocker A^T ?

        }
        __syncthreads();
    }
    for (int iter = N - 1; iter >= 0; iter--) {
        if (index == 0) {
            ytemp[iter] = ytemp[iter] / A[iter * N + iter];
            y[iter] = ytemp[iter];
        }
        __syncthreads();

        for (int i = index; i < iter; i += step)
        {
            ytemp[i] = ytemp[i] - ytemp[iter] * A[i * N + iter]; // moche ne faudrait-il pas stocker A^T ?

        }
        __syncthreads();
    }
}
