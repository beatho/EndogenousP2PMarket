#include "hip/hip_runtime.h"


#include "../head/TestUtilities.cuh"




int testUtilities() {
	int n = 1;
	if (!testcoefPolynome3From4to2coef1()) return n;
	n++;
	if (!testcoefPolynome3From4to2coef2()) return n;
	n++;
	if (!testcoefPolynome3From4to2coef3()) return n;
	n++;
	if (!testresolveRealPolynome3without2term1()) return n;
	n++;
	if (!testresolveRealPolynome3without2term2()) return n;
	n++;
	if (!testresolveRealPolynome3without2term3()) return n;
	n++;
	if (!testresolveRealPolynome4without2term()) return n;
	n++;
	if (!testresolveRealPolynome4without2term2()) return n;
	n++;
	if (!testresolveRealPolynome4without2termLagrange()) return n;
	n++;
	if (!testresolveRealPolynome4without2term2Lagrange()) return n;
		n++;
	std::cout << "---------- GPU ------------------" << std::endl;
	if (!testresolveRealPolynome3without2termGPU()) return n;
	n++; 
	if (!testresolveRealPolynome4without2termGPU()) return n;
	n++;
	if (!testresolveRealPolynome4without2termGPULagrange()) return n;
	n++;
	std::cout << "---------- Eigen------------------" << std::endl;
	if (!testPolyEigen3()) return n;
	n++;
	if (!testPolyEigen4()) return n;
	n++;
	if (!testresolveRealPolynome3without2termEigen()) return n;
	n++;
	if (!testresolveRealPolynome4without2termEigen()) return n;
	n++;
	std::cout << "---------- NEWTON------------------" << std::endl;
	if (!testresolveRealPolynome3Newton1()) return n;
	n++;
	if (!testresolveRealPolynome3Newton2()) return n;
	n++;
	if (!testresolveRealPolynome3Newton3()) return n;
	n++;
	if (!testresolveRealPolynome4Newton1()) return n;
	n++;
	if (!testresolveRealPolynome4Newton2()) return n;
	n++; 
	std::cout << "---------- Halley ------------------" << std::endl;
	if (!testresolveRealPolynome3Halley1()) return n;
	n++;
	if (!testresolveRealPolynome3Halley2()) return n;
	n++;
	if (!testresolveRealPolynome3Halley3()) return n;
	n++;
	if (!testresolveRealPolynome4Halley1()) return n;
	n++;
	if (!testresolveRealPolynome4Halley2()) return n;
	n++; // 20
	std::cout << "---------- GPU 2------------------" << std::endl;
	if (!testresolveRealPolynome3GPU()) return n;
	n++;  
	if (!testresolveRealPolynome4GPU()) return n;
	n++;
	std::cout << "---------- Laguerre------------------" << std::endl;
	if (!testresolveRealPolynome3Laguerre1()) return n;
	n++;
	if (!testresolveRealPolynome3Laguerre2()) return n;
	n++;
	if (!testresolveRealPolynome3Laguerre3()) return n;
	n++;

	return 0;
}

void compareCPUGPU()
{
	std::string fileName = "rootCPUGPU2.csv";


	std::chrono::high_resolution_clock::time_point t1;
	std::chrono::high_resolution_clock::time_point t2;
	


	// faire varier p et q
	double qmax = 100;
	double qmin = 0;
	double pmax = 100;
	double pmin = -100;
	int nQ = 100;
	int nP = 200;

	int blocksize = 512;

	int N = nQ * nP; // nombre total de polynome � r�soudre

	int Numblock = ceil((N + blocksize - 1) / blocksize);

	MatrixCPU Param(1, 10);
	Param.set(0, 0, nQ);
	Param.set(0, 1, nP);
	Param.set(0, 2, qmin);
	Param.set(0, 3, qmax);
	Param.set(0, 4, pmin);
	Param.set(0, 5, pmax);
	Param.set(0, 6, blocksize);
	Param.set(0, 7, Numblock);


	// calcul des pas
	double dP = (pmax - pmin) / nP;
	double dQ = (qmax - qmin) / nQ;

	MatrixGPUD coef(4, N);
	MatrixGPUD rootsGPU(3, N, nanf(""), 1);
	MatrixCPU  rootsCPU(3, N, nanf(""));
	MatrixCPU nRootCPU(1, N);
	MatrixGPUD nRootGPU(1, N, 0, 1);

	int poly = 0;
	for (int i = 0; i < nP; i++) {
		float p = pmin + i * dP;
		for (int j = 0; j < nQ; j++) {
			float q = qmin + j * dQ;
			coef.set(0, poly, 1);
			coef.set(1, poly, 0);
			coef.set(2, poly, p);
			coef.set(3, poly, q);
			poly++;
		}
	}
	
	// resolution CPU
	double coef2[2];
	double root[3];
	t1 = std::chrono::high_resolution_clock::now();
	for (int n = 0; n < N; n++) {
		coef2[0] = coef.get(2, n);
		coef2[1] = coef.get(3, n);
		int nRoot = resolveRealPolynome3without2term(root, coef2);
		nRootCPU.set(0, n, nRoot);
		for (int i = 0; i < nRoot; i++) {
			rootsCPU.set(i, n, root[i]);
		}
	}
	t2 = std::chrono::high_resolution_clock::now();
	
	Param.set(0, 8, (float)std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count()/ BILLION);

	//resolution GPU

	t1 = std::chrono::high_resolution_clock::now();
	coef.transferGPU();
	resolveSeveralRealPolynome3termGPU << <Numblock, blocksize >> > (nRootGPU._matrixGPU, rootsGPU._matrixGPU, coef._matrixGPU, N);
	hipDeviceSynchronize();
	t2 = std::chrono::high_resolution_clock::now();


	Param.set(0, 9, (float)std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count() / BILLION);
	coef.transferCPU();
	rootsGPU.transferCPU();
	nRootGPU.transferCPU();
	//save

	Param.saveCSV(fileName);
	coef.saveCSV(fileName);
	nRootCPU.saveCSV(fileName);
	rootsCPU.saveCSV(fileName);
	nRootGPU.saveCSV(fileName);
	rootsGPU.saveCSV(fileName);




}

bool testcoefPolynome3From4to2coef1()
{
	double coef4[4] = { 1.5, 2.2, -3.5, -4 };
	double coef2[2];

	double pSol = -3.0504;
	double qSol = -1.2922;
	coefPolynome3From4to2coef(coef4, coef2);

	if (abs(pSol - coef2[0])>0.001 || abs(qSol - coef2[1])>0.001) {
		std::cout << "p " << pSol << " " << coef2[0] << " q " << qSol << " " << coef2[1] << std::endl;
		return false;
	}

	return true;
}

bool testcoefPolynome3From4to2coef2()
{
	double coef4[4] = { 0, 2.2, -3.5, -4 };
	double coef2[2];
	try
	{
		coefPolynome3From4to2coef(coef4, coef2);
	}
	catch (const std::exception&)
	{
		return true;
	}

	return false;
}

bool testcoefPolynome3From4to2coef3()
{
	double coef4[4] = { 1, 0, -3.5, -4 };
	double coef2[2];

	double pSol = -3.5;
	double qSol = -4;
	coefPolynome3From4to2coef(coef4, coef2);

	if (abs(pSol - coef2[0]) > 0.001 || abs(qSol - coef2[1]) > 0.001) {
		std::cout << "p " << pSol << " " << coef2[0] << " q " << qSol << " " << coef2[1] << std::endl;
		return false;
	}

	return true;
}

bool testresolveRealPolynome3without2term1() {

	double a = 1;
	double b = -5;
	
	double coef4[4] = { 1, -5, 3, 1 };
	double coef2[2];
	double root1 = 2 - sqrt(5);
	double root2 = 1;
	double root3 = 2 + sqrt(5);

	coefPolynome3From4to2coef(coef4, coef2);
	double root[3];

	int nRoot = resolveRealPolynome3without2term(root, coef2);
	if (nRoot != 3) {
		std::cout << "wrong number of root " << nRoot << std::endl;
		return false;
	}
	for (int k = 0; k < 3; k++) {
		root[k] += -b / (3 * a);
	}

	bool find[3] = { false, false, false };
	for (int k = 0; k < 3; k++) {
		
		if (abs(root[k] - root1)<0.001) {
			find[0] = true;
		}
		else if (abs(root[k] - root2) < 0.001) {
			find[1] = true;
		}
		else if (abs(root[k] - root3) < 0.001) {
			find[2] = true;
		}
		else {
			std::cout << "wrong root " << root[k] << " against " << root1 << " " << root2 << " " << root3 << std::endl;
			return false;
		}
	}

	for (int k = 0; k < 3; k++) {
		if (!find[k]) {
			std::cout << "wrong root " << root[0] << " " << root[1] << " " << root[2] << " against " << root1 << " " << root2 << " " << root3 << std::endl;
			return false;
		}
	}


	return true;
}
bool testresolveRealPolynome3without2term2() {
	double a = 1;
	double b = 0;
	double c = 3;
	double d = 1;

	double coef4[4] = { 1, 0, 3, 1 };
	double coef2[2];
	double root1 = cbrt((-1 + sqrt(5)) / 2) + cbrt((-1 - sqrt(5)) / 2);
	
	coefPolynome3From4to2coef(coef4, coef2);
	double root[1];

	int nRoot = resolveRealPolynome3without2term(root, coef2);
	if (nRoot != 1) {
		std::cout << "wrong number of root " << nRoot << std::endl;
		return false;
	}
	
	root[0] += -b / (3 * a);
	
	
	if (abs(root[0] - root1) > 0.001) {
		std::cout << "wrong root " << root[0] << " against " << root1 << std::endl;
		return false;
	}
		

	return true;
}
bool testresolveRealPolynome3without2term3() {
	double a = 1;
	double b = 2;
	double c = -12.75;
	double d = 11.25;

	double coef4[4] = { 1, 2,  -12.75, 11.25 };
	double coef2[2];
	double root1 = 1.5;
	double root2 = -5;
	

	coefPolynome3From4to2coef(coef4, coef2);
	
	std::cout << "poly 3 " << coef2[0] << " " << coef2[1] << std::endl;
	
	double root[3];

	int nRoot = resolveRealPolynome3without2term(root, coef2);
	if (nRoot == 1) {
		std::cout << "wrong number of root " << nRoot << std::endl;
		return false;
	}
	for (int k = 0; k < 2; k++) {
		root[k] += -b / (3 * a);
	}

	bool find[2] = { false, false };
	for (int k = 0; k < 2; k++) {

		if (abs(root[k] - root1) < 0.001) {
			find[0] = true;
		}
		else if (abs(root[k] - root2) < 0.001) {
			find[1] = true;
		}
		else {
			std::cout << "wrong root " << root[k] << " against " << root1 << " " << root2 << " " << std::endl;
			return false;
		}
	}

	for (int k = 0; k < 2; k++) {
		if (!find[k]) {
			std::cout << "wrong root " << root[0] << " " << root[1] << " against " << root1 << " " << root2 << std::endl;
			return false;
		}
	}
	return true;
}


bool testresolveRealPolynome4without2term()
{
	double rootbis[3];
	double coef4[4] = { 1, 7, 7, -6 };
	double coef3[3] = { 6, -13, 6 };
	double coef2[2];
	coefPolynome3From4to2coef(coef4, coef2);
	int nroot = resolveRealPolynome3without2term(rootbis, coef2);

	for (int k = 0; k < nroot; k++) {
		rootbis[k] += -coef4[1] / (3 * coef4[0]);
	}

	double root1 = rootbis[0];
	double root2 = rootbis[1];
	double root3 = rootbis[2];
	double root4 = 1;

	double root[4];

	int nRoot = resvolveRealPolynome4without2term(root, coef3);
	if (nRoot != 4) {
		std::cout << "wrong number of root " << nRoot << std::endl;
		return false;
	}
	
	bool find[4] = { false, false, false, false};
	for (int k = 0; k < 4; k++) { // si racine multiple ne renvoie pas d'erreur si on ne trouve pas la bonne multiplicit� mais les bonnes racines -> pas grave dans notre cas.

		if (abs(root[k] - root1) < 0.001) {
			find[0] = true;
		}
		else if (abs(root[k] - root2) < 0.001) {
			find[1] = true;
		}
		else if (abs(root[k] - root3) < 0.001) {
			find[2] = true;
		}
		else if (abs(root[k] - root4) < 0.001) {
			find[3] = true;
		}
		else {
			std::cout << "wrong root " << root[k] << " against " << root1 << " " << root2 << " " << root3 << " " << root4 << std::endl;
			return false;
		}
	}

	for (int k = 0; k < 4; k++) {
		if (!find[k]) {
			std::cout << "wrong root " << root[0] << " " << root[1] << " " << root[2] << " " << root[3] << " against " << root1 << " " << root2 << " " << root3 << " " << root4 << std::endl;
			return false;
		}
	}


	return true;
}
bool testresolveRealPolynome4without2term2()
{
	double root[4];
	double coef3[3] = { -109.778, -4260.6, -3051.76 };
	
	
	int nRoot = resvolveRealPolynome4without2term(root, coef3);
	if (nRoot != 2) {
		std::cout << "wrong number of root " << nRoot << std::endl;
		return false;
	}
	double rootBis[2] = { -0.707106, 110.132 };

	for (int i = 0; i < nRoot; i++) {
		double r = root[i];
		double poly = coef3[2] + coef3[1] * r + coef3[0] * r * r * r + r * r * r * r;
		if (abs(poly) > 0.000001) {
			std::cout << "wrong root poly= "<< poly << " " << root[0] << " " << root[1] << " against " << rootBis[0] << " " << rootBis[1] << std::endl;
			return false;
		}
	}

	return true;
}


bool testresolveRealPolynome4without2termLagrange()
{
	double rootbis[3];
	double coef4[4] = { 1, 7, 7, -6 };
	double coef3[3] = { 6, -13, 6 };
	double coef2[2];
	coefPolynome3From4to2coef(coef4, coef2);
	int nroot = resolveRealPolynome3without2term(rootbis, coef2);

	for (int k = 0; k < nroot; k++) {
		rootbis[k] += -coef4[1] / (3 * coef4[0]);
	}

	double root1 = rootbis[0];
	double root2 = rootbis[1];
	double root3 = rootbis[2];
	double root4 = 1;

	double root[4];

	int nRoot = resvolveRealPolynome4without2termLagrange(root, coef3);
	if (nRoot != 4) {
		std::cout << "wrong number of root " << nRoot << std::endl;
		return false;
	}

	bool find[4] = { false, false, false, false };
	for (int k = 0; k < 4; k++) { // si racine multiple ne renvoie pas d'erreur si on ne trouve pas la bonne multiplicit� mais les bonnes racines -> pas grave dans notre cas.

		if (abs(root[k] - root1) < 0.001) {
			find[0] = true;
		}
		else if (abs(root[k] - root2) < 0.001) {
			find[1] = true;
		}
		else if (abs(root[k] - root3) < 0.001) {
			find[2] = true;
		}
		else if (abs(root[k] - root4) < 0.001) {
			find[3] = true;
		}
		else {
			std::cout << "wrong root " << root[k] << " against " << root1 << " " << root2 << " " << root3 << " " << root4 << std::endl;
			return false;
		}
	}

	for (int k = 0; k < 4; k++) {
		if (!find[k]) {
			std::cout << "wrong root " << root[0] << " " << root[1] << " " << root[2] << " " << root[3] << " against " << root1 << " " << root2 << " " << root3 << " " << root4 << std::endl;
			return false;
		}
	}


	return true;
}
bool testresolveRealPolynome4without2term2Lagrange()
{
	double root[4];
	double coef3[3] = { -109.778, -4260.6, -3051.76 };


	int nRoot = resvolveRealPolynome4without2termLagrange(root, coef3);
	if (nRoot != 2) {
		std::cout << "wrong number of root " << nRoot << std::endl;
		return false;
	}
	double rootBis[2] = { -0.707106, 110.132 };

	for (int i = 0; i < nRoot; i++) {
		double r = root[i];
		double poly = coef3[2] + coef3[1] * r + coef3[0] * r * r * r + r * r * r * r;
		if (abs(poly) > 0.000001) {
			std::cout << "wrong root poly= " << poly << " " << root[0] << " " << root[1] << " against " << rootBis[0] << " " << rootBis[1] << std::endl;
			return false;
		}
	}

	return true;
}

bool testresolveRealPolynome3without2termGPU() {
	
	int nPoly = 3;
	MatrixGPUD coefs(4, nPoly);
	MatrixGPUD roots(3, nPoly, 0, 1);
	MatrixGPUD rootToFind(3, nPoly);
	MatrixGPUD nRoot(nPoly, 1, 0, 1);
	MatrixGPUD nRootToFind(nPoly, 1);

	int poly = 0;
	// --------poly 1-----------
	//double coef4[4] = { 1, -5, 3, 1 };
	coefs.set(0, poly, 1);
	coefs.set(1, poly, -5);
	coefs.set(2, poly, 3);
	coefs.set(3, poly, 1);
	// double root1 = 2 - sqrt(5); 	double root2 = 1; 	double root3 = 2 + sqrt(5);
	rootToFind.set(0, poly, 2 + sqrt(5));
	rootToFind.set(1, poly, 2 - sqrt(5));
	rootToFind.set(2, poly, 1);
	
	nRootToFind.set(poly, 0, 3);
	poly++;
	//--------poly 2-----------
	//double coef4[4] = { 1, 0, 3, 1 };
	coefs.set(0, poly, 1);
	coefs.set(1, poly, 0);
	coefs.set(2, poly, 3);
	coefs.set(3, poly, 1);
	// root1 = cbrt((-1 + sqrt(5)) / 2) + cbrt((-1 - sqrt(5)) / 2);
	rootToFind.set(0, poly, cbrt((-1 + sqrt(5)) / 2) + cbrt((-1 - sqrt(5)) / 2));
	nRootToFind.set(poly, 0, 1);
	poly++;


	// poly 3
	//double coef4[4] = { 1, 2,  -12.75, 11.25 };
	coefs.set(0, poly, 1);
	coefs.set(1, poly, 2);
	coefs.set(2, poly, -12.75);
	coefs.set(3, poly, 11.25);
	// double root1 = 1.5; 	double root2 = -5;
	rootToFind.set(0, poly, 1.5);
	rootToFind.set(1, poly, -5);
	nRootToFind.set(poly, 0, 2);
	poly++;

	coefs.transferGPU();

	resolveSeveralRealPolynome3termGPU << <1, 32 >> > (nRoot._matrixGPU, roots._matrixGPU, coefs._matrixGPU, nPoly);

	nRoot.transferCPU();
	roots.transferCPU();

	nRoot.display();
	nRootToFind.display();

	roots.display();
	rootToFind.display();

	return true;


}
bool testresolveRealPolynome4without2termGPU() {

	int nPoly = 2;
	MatrixGPUD coefs(4, nPoly);
	MatrixGPUD roots(4, nPoly, 0, 1);
	MatrixGPUD rootToFind(4, nPoly);
	MatrixGPUD nRoot(nPoly, 1, 0, 1);
	MatrixGPUD nRootToFind(nPoly, 1);

	int poly = 0;
	// --------poly 1-----------
	// double coef3[3] = { 6, -13, 6 };
		// determination des racines
	double rootbis[3];
	double coef4[4] = { 1, 7, 7, -6 };
	double coef2[2];
	coefPolynome3From4to2coef(coef4, coef2);
	int nroot = resolveRealPolynome3without2term(rootbis, coef2);

	coefs.set(0, poly, 6);
	coefs.set(2, poly, -13);
	coefs.set(3, poly, 6);

	for (int k = 0; k < nroot; k++) {
		rootbis[k] += -coef4[1] / (3 * coef4[0]);
	}
	rootToFind.set(0, poly, rootbis[2]);
	rootToFind.set(1, poly, rootbis[1]);
	rootToFind.set(2, poly, 1);
	rootToFind.set(3, poly, rootbis[0]);

	nRootToFind.set(poly, 0, 4);
	poly++;

	// --------poly 2-----------
// double coef3[3] = { -109.778, -4260.6, -3051.76 };

	coefs.set(0, poly, -109.778);
	coefs.set(2, poly, -4260.6);
	coefs.set(3, poly, -3051.76);	
	rootToFind.set(0, poly, 110.132);
	rootToFind.set(1, poly, -0.707106);


	nRootToFind.set(poly, 0, 2);
	coefs.transferGPU();

	resolveSeveralRealPolynome4WO2termGPU << <1, 32 >> > (nRoot._matrixGPU, roots._matrixGPU, coefs._matrixGPU, nPoly);

	nRoot.transferCPU();
	roots.transferCPU();

	nRoot.display();
	nRootToFind.display();

	roots.display();
	rootToFind.display();

	return true;
}

bool testresolveRealPolynome4without2termGPULagrange() {

	int nPoly = 2;
	MatrixGPUD coefs(4, nPoly);
	MatrixGPUD roots(4, nPoly, 0, 1);
	MatrixGPUD rootToFind(4, nPoly);
	MatrixGPUD nRoot(nPoly, 1, 0, 1);
	MatrixGPUD nRootToFind(nPoly, 1);

	int poly = 0;
	// --------poly 1-----------
	// double coef3[3] = { 6, -13, 6 };
		// determination des racines
	double rootbis[3];
	double coef4[4] = { 1, 7, 7, -6 };
	double coef2[2];
	coefPolynome3From4to2coef(coef4, coef2);
	int nroot = resolveRealPolynome3without2term(rootbis, coef2);

	coefs.set(0, poly, 6);
	coefs.set(2, poly, -13);
	coefs.set(3, poly, 6);

	for (int k = 0; k < nroot; k++) {
		rootbis[k] += -coef4[1] / (3 * coef4[0]);
	}
	rootToFind.set(0, poly, rootbis[2]);
	rootToFind.set(1, poly, rootbis[1]);
	rootToFind.set(2, poly, 1);
	rootToFind.set(3, poly, rootbis[0]);

	nRootToFind.set(poly, 0, 4);
	poly++;

	// --------poly 2-----------
// double coef3[3] = { -109.778, -4260.6, -3051.76 };

	coefs.set(0, poly, -109.778);
	coefs.set(2, poly, -4260.6);
	coefs.set(3, poly, -3051.76);
	rootToFind.set(0, poly, 110.132);
	rootToFind.set(1, poly, -0.707106);


	nRootToFind.set(poly, 0, 2);
	coefs.transferGPU();

	resolveSeveralRealPolynome4WO2termGPULagrange << <1, 32 >> > (nRoot._matrixGPU, roots._matrixGPU, coefs._matrixGPU, nPoly);

	nRoot.transferCPU();
	roots.transferCPU();

	nRoot.display();
	nRootToFind.display();

	roots.display();
	rootToFind.display();

	return true;
}

bool testPolyEigen3()
{
	Eigen::Vector4d coeff(1, 3, -5, 1); //double coef4[4] = { 1, -5, 3, 1 };
	Eigen::PolynomialSolver<double, Eigen::Dynamic> solver;
	solver.compute(coeff);
	const Eigen::PolynomialSolver<double, Eigen::Dynamic>::RootsType& r = solver.roots();

	std::cout << r << std::endl;
	
	double root1 = 2 - sqrt(5);
	double root2 = 1;
	double root3 = 2 + sqrt(5);
	double root[3];

	if (r(0).imag() || r(1).imag() || r(2).imag()) {
		std::cout << r(0).imag() << " " << r(1).imag() << " " << r(2).imag() << std::endl;
		return false;
	}
	bool find[3] = { false, false, false };
	
	for (int k = 0; k < 3; k++) {
		root[k] = r(k).real();
	}
	
	for (int k = 0; k < 3; k++) {

		if (abs(root[k] - root1) < 0.001) {
			find[0] = true;
		}
		else if (abs(root[k] - root2) < 0.001) {
			find[1] = true;
		}
		else if (abs(root[k] - root3) < 0.001) {
			find[2] = true;
		}
		else {
			std::cout << "wrong root " << root[k] << " against " << root1 << " " << root2 << " " << root3 << std::endl;
			return false;
		}
	}

	for (int k = 0; k < 3; k++) {
		if (!find[k]) {
			std::cout << "wrong root " << root[0] << " " << root[1] << " " << root[2] << " against " << root1 << " " << root2 << " " << root3 << std::endl;
			return false;
		}
	}
	std::cout << std::endl << std::endl;

	return true;
}

bool testPolyEigen4()
{
	Eigen::VectorXd coeff(5);
	coeff(0) = 6;
	coeff(1) = -13;
	coeff(2) = 0;
	coeff(3) = 6;
	coeff(4) = 1;
	
	Eigen::PolynomialSolver<double, Eigen::Dynamic> solver;
	solver.compute(coeff);
	const Eigen::PolynomialSolver<double, Eigen::Dynamic>::RootsType& r = solver.roots();

	std::cout << r << std::endl;

	double rootbis[3];
	double coef4[4] = { 1, 7, 7, -6 };
	double coef3[3] = { 6, -13, 6 };
	double coef2[2];
	coefPolynome3From4to2coef(coef4, coef2);
	int nroot = resolveRealPolynome3without2term(rootbis, coef2);

	for (int k = 0; k < nroot; k++) {
		rootbis[k] += -coef4[1] / (3 * coef4[0]);
	}

	double root1 = rootbis[0];
	double root2 = rootbis[1];
	double root3 = rootbis[2];
	double root4 = 1;

	double root[4];

	bool find[4] = { false, false, false, false };

	for (int k = 0; k < 4; k++) {
		if (r(k).imag()) {
			std::cout << r(k).imag() << std::endl;
			return false;
		}
		root[k] = r(k).real();
	}


	for (int k = 0; k < 4; k++) { // si racine multiple ne renvoie pas d'erreur si on ne trouve pas la bonne multiplicit� mais les bonnes racines -> pas grave dans notre cas.

		if (abs(root[k] - root1) < 0.001) {
			find[0] = true;
		}
		else if (abs(root[k] - root2) < 0.001) {
			find[1] = true;
		}
		else if (abs(root[k] - root3) < 0.001) {
			find[2] = true;
		}
		else if (abs(root[k] - root4) < 0.001) {
			find[3] = true;
		}
		else {
			std::cout << "wrong root " << root[k] << " against " << root1 << " " << root2 << " " << root3 << " " << root4 << std::endl;
			return false;
		}
	}

	for (int k = 0; k < 4; k++) {
		if (!find[k]) {
			std::cout << "wrong root " << root[0] << " " << root[1] << " " << root[2] << " " << root[3] << " against " << root1 << " " << root2 << " " << root3 << " " << root4 << std::endl;
			return false;
		}
	}

	std::cout << std::endl << std::endl;

	return true;
}

bool testresolveRealPolynome3without2termEigen()
{
	{
		double a = 1;
		double b = -5;

		double coef4[4] = { 1, -5, 3, 1 };
		double coef2[2];
		double root1 = 2 - sqrt(5);
		double root2 = 1;
		double root3 = 2 + sqrt(5);

		coefPolynome3From4to2coef(coef4, coef2);

		
		double root[3];

		int nRoot = resolveRealPolynome3without2termEigen(root, coef2);
		if (nRoot != 3) {
			std::cout << "wrong number of root " << nRoot << std::endl;
			return false;
		}
		for (int k = 0; k < 3; k++) {
			root[k] += -b / (3 * a);
		}

		bool find[3] = { false, false, false };
		for (int k = 0; k < 3; k++) {

			if (abs(root[k] - root1) < 0.001) {
				find[0] = true;
			}
			else if (abs(root[k] - root2) < 0.001) {
				find[1] = true;
			}
			else if (abs(root[k] - root3) < 0.001) {
				find[2] = true;
			}
			else {
				std::cout << "wrong root " << root[k] << " against " << root1 << " " << root2 << " " << root3 << std::endl;
				return false;
			}
		}

		for (int k = 0; k < 3; k++) {
			if (!find[k]) {
				std::cout << "wrong root " << root[0] << " " << root[1] << " " << root[2] << " against " << root1 << " " << root2 << " " << root3 << std::endl;
				return false;
			}
		}
		std::cout << "root Eigen ";
		for (int k = 0; k < nRoot; k++) {

			std::cout << root[k] << " ";
		}
		std::cout << std::endl;
	}

	{
		double a = 1;
		double b = 0;
		double c = 3;
		double d = 1;

		double coef4[4] = { 1, 0, 3, 1 };
		double coef2[2];
		double root1 = cbrt((-1 + sqrt(5)) / 2) + cbrt((-1 - sqrt(5)) / 2);

		coefPolynome3From4to2coef(coef4, coef2);
		double root[1];

		int nRoot = resolveRealPolynome3without2term(root, coef2);
		if (nRoot != 1) {
			std::cout << "wrong number of root " << nRoot << std::endl;
			return false;
		}

		root[0] += -b / (3 * a);


		if (abs(root[0] - root1) > 0.001) {
			std::cout << "wrong root " << root[0] << " against " << root1 << std::endl;
			return false;
		}
		std::cout << "root Eigen ";
		for (int k = 0; k < nRoot; k++) {

			std::cout << root[k] << " ";
		}
		std::cout << std::endl;
	}


	{
		double a = 1;
		double b = 2;
		double c = -12.75;
		double d = 11.25;

		double coef4[4] = { 1, 2,  -12.75, 11.25 };
		double coef2[2];
		double root1 = 1.5;
		double root2 = -5;


		coefPolynome3From4to2coef(coef4, coef2);

		std::cout << "poly 3 " << coef2[0] << " " << coef2[1] << std::endl;

		double root[3];

		int nRoot = resolveRealPolynome3without2term(root, coef2);
		if (nRoot == 1) {
			std::cout << "wrong number of root " << nRoot << std::endl;
			return false;
		}
		for (int k = 0; k < 2; k++) {
			root[k] += -b / (3 * a);
		}

		bool find[2] = { false, false };
		for (int k = 0; k < 2; k++) {

			if (abs(root[k] - root1) < 0.001) {
				find[0] = true;
			}
			else if (abs(root[k] - root2) < 0.001) {
				find[1] = true;
			}
			else {
				std::cout << "wrong root " << root[k] << " against " << root1 << " " << root2 << " " << std::endl;
				return false;
			}
		}

		for (int k = 0; k < 2; k++) {
			if (!find[k]) {
				std::cout << "wrong root " << root[0] << " " << root[1] << " against " << root1 << " " << root2 << std::endl;
				return false;
			}
		}
		std::cout << "root Eigen ";
		for (int k = 0; k < nRoot; k++) {

			std::cout << root[k] << " ";
		}
		std::cout << std::endl;
	}
	return true;
}


bool testresolveRealPolynome4without2termEigen()
{
	{
		double rootbis[3];
		double coef4[4] = { 1, 7, 7, -6 };
		double coef3[3] = { 6, -13, 6 };
		double coef2[2];
		coefPolynome3From4to2coef(coef4, coef2);
		int nroot = resolveRealPolynome3without2termEigen(rootbis, coef2);

		for (int k = 0; k < nroot; k++) {
			rootbis[k] += -coef4[1] / (3 * coef4[0]);
		}

		double root1 = rootbis[0];
		double root2 = rootbis[1];
		double root3 = rootbis[2];
		double root4 = 1;

		double root[4];

		int nRoot = resvolveRealPolynome4without2termEigen(root, coef3);
		if (nRoot != 4) {
			std::cout << "wrong number of root " << nRoot << std::endl;
			return false;
		}

		bool find[4] = { false, false, false, false };
		for (int k = 0; k < 4; k++) { // si racine multiple ne renvoie pas d'erreur si on ne trouve pas la bonne multiplicit� mais les bonnes racines -> pas grave dans notre cas.

			if (abs(root[k] - root1) < 0.001) {
				find[0] = true;
			}
			else if (abs(root[k] - root2) < 0.001) {
				find[1] = true;
			}
			else if (abs(root[k] - root3) < 0.001) {
				find[2] = true;
			}
			else if (abs(root[k] - root4) < 0.001) {
				find[3] = true;
			}
			else {
				std::cout << "wrong root " << root[k] << " against " << root1 << " " << root2 << " " << root3 << " " << root4 << std::endl;
				return false;
			}
		}

		for (int k = 0; k < 4; k++) {
			if (!find[k]) {
				std::cout << "wrong root " << root[0] << " " << root[1] << " " << root[2] << " " << root[3] << " against " << root1 << " " << root2 << " " << root3 << " " << root4 << std::endl;
				return false;
			}
		}
		std::cout << "root Eigen ";
		for (int k = 0; k < nRoot; k++) {
			
			std::cout << root[k] << " ";
		}
		std::cout << std::endl;
	}


	{
		double root[4];
		double coef3[3] = { -109.778, -4260.6, -3051.76 };


		int nRoot = resvolveRealPolynome4without2termEigen(root, coef3);
		if (nRoot != 2) {
			std::cout << "wrong number of root " << nRoot << std::endl;
			return false;
		}
		double rootBis[2] = { -0.707106, 110.132 };

		for (int i = 0; i < nRoot; i++) {
			double r = root[i];
			double poly = coef3[2] + coef3[1] * r + coef3[0] * r * r * r + r * r * r * r;
			if (abs(poly) > 0.000001) {
				std::cout << "wrong root poly= " << poly << " " << root[0] << " " << root[1] << " against " << rootBis[0] << " " << rootBis[1] << std::endl;
				return false;
			}
		}

		std::cout << "root Eigen ";
		for (int k = 0; k < nRoot; k++) {

			std::cout << root[k] << " ";
		}
		std::cout << std::endl;
	}
	return true;
}


bool testresolveRealPolynome3Newton1() {

	double a = 1;
	double b = -5;

	double coef3[3] = {-5, 3, 1 };
	double root1 = 2 - sqrt(5);
	double root2 = 1;
	double root3 = 2 + sqrt(5);

	double root[3];

	int nRoot = resolveRealPolynome3Newton(root, coef3);
	if (nRoot != 3) {
		std::cout << "wrong number of root " << nRoot << std::endl;
		return false;
	}
	
	bool find[3] = { false, false, false };
	for (int k = 0; k < 3; k++) {

		if (abs(root[k] - root1) < 0.001) {
			find[0] = true;
		}
		else if (abs(root[k] - root2) < 0.001) {
			find[1] = true;
		}
		else if (abs(root[k] - root3) < 0.001) {
			find[2] = true;
		}
		else {
			std::cout << "wrong root " << root[k] << " against " << root1 << " " << root2 << " " << root3 << std::endl;
			return false;
		}
	}

	for (int k = 0; k < 3; k++) {
		if (!find[k]) {
			std::cout << "wrong root " << root[0] << " " << root[1] << " " << root[2] << " against " << root1 << " " << root2 << " " << root3 << std::endl;
			return false;
		}
	}


	return true;
}
bool testresolveRealPolynome3Newton2() {
	
	double a = 1;
	double b = 0;
	double c = 3;
	double d = 1;

	double coef3[3] = { 0, 3, 1 };
	double root1 = cbrt((-1 + sqrt(5)) / 2) + cbrt((-1 - sqrt(5)) / 2);


	double root[3];

	int nRoot = resolveRealPolynome3Newton(root, coef3);
	
	if (nRoot != 1) {
		std::cout << "wrong number of root " << nRoot << std::endl;
		return false;
	}



	if (abs(root[0] - root1) > 0.001) {
		std::cout << "wrong root " << root[0] << " against " << root1 << std::endl;
		return false;
	}


	return true;
}
bool testresolveRealPolynome3Newton3() {
	double a = 1;
	double b = 2;
	double c = -12.75;
	double d = 11.25;

	double coef3[3] = { 2,  -12.75, 11.25 };
	
	double root1 = 1.5;
	double root2 = -5;



	double root[3];

	int nRoot = resolveRealPolynome3Newton(root, coef3);
	if (nRoot == 1) {
		std::cout << "wrong number of root " << nRoot << std::endl;
		return false;
	}
	
	bool find[2] = { false, false };
	for (int k = 0; k < nRoot; k++) {

		if (abs(root[k] - root1) < 0.001) {
			find[0] = true;
		}
		else if (abs(root[k] - root2) < 0.001) {
			find[1] = true;
		}
		else {
			std::cout << "wrong root " << root[k] << " against " << root1 << " " << root2 << " " << std::endl;
			return false;
		}
	}

	for (int k = 0; k < 2; k++) {
		if (!find[k]) {
			std::cout << "wrong root " << root[0] << " " << root[1] << " against " << root1 << " " << root2 << std::endl;
			return false;
		}
	}
	return true;
}



bool testresolveRealPolynome4Newton1()
{
	double rootbis[3];
	double coef4[4] = { 1, 7, 7, -6 }; //poly 3 avec tous les coefs
	
	double coef4bis[4] = { 6, 0, -13, 6 }; // poly 4 avec les coef sans le premier unitaire

	// juste pour trouver les racines
	double coef2[2];
	coefPolynome3From4to2coef(coef4, coef2);
	int nroot = resolveRealPolynome3without2term(rootbis, coef2);

	for (int k = 0; k < nroot; k++) {
		rootbis[k] += -coef4[1] / (3 * coef4[0]);
	}

	double root1 = rootbis[0];
	double root2 = rootbis[1];
	double root3 = rootbis[2];
	double root4 = 1;


	// vrai test
	double root[4];

	int nRoot = resolveRealPolynome4Newton(root, coef4bis);
	if (nRoot != 4) {
		std::cout << "wrong number of root " << nRoot << std::endl;
		return false;
	}

	bool find[4] = { false, false, false, false };
	for (int k = 0; k < 4; k++) { // si racine multiple ne renvoie pas d'erreur si on ne trouve pas la bonne multiplicit� mais les bonnes racines -> pas grave dans notre cas.

		if (abs(root[k] - root1) < 0.001) {
			find[0] = true;
		}
		else if (abs(root[k] - root2) < 0.001) {
			find[1] = true;
		}
		else if (abs(root[k] - root3) < 0.001) {
			find[2] = true;
		}
		else if (abs(root[k] - root4) < 0.001) {
			find[3] = true;
		}
		else {
			std::cout << "wrong root " << root[k] << " against " << root1 << " " << root2 << " " << root3 << " " << root4 << std::endl;
			return false;
		}
	}

	for (int k = 0; k < 4; k++) {
		if (!find[k]) {
			std::cout << "wrong root " << root[0] << " " << root[1] << " " << root[2] << " " << root[3] << " against " << root1 << " " << root2 << " " << root3 << " " << root4 << std::endl;
			return false;
		}
	}


	return true;
}
bool testresolveRealPolynome4Newton2()
{
	double root[4];
	double coef4[4] = { -109.778, 0, -4260.6, -3051.76 };


	int nRoot = resolveRealPolynome4Newton(root, coef4);
	if (nRoot != 2) {
		std::cout << "wrong number of root " << nRoot << std::endl;
		return false;
	}
	double rootBis[2] = { -0.707106, 110.132 };

	for (int i = 0; i < nRoot; i++) {
		double r = root[i];
		double poly = coef4[3] + coef4[2] * r + coef4[0] * r * r * r + r * r * r * r;
		if (abs(poly) > 0.000001) {
			std::cout << "wrong root poly= " << poly << " " << root[0] << " " << root[1] << " against " << rootBis[0] << " " << rootBis[1] << std::endl;
			return false;
		}
	}

	return true;
}



bool testresolveRealPolynome3Laguerre1() {

	double a = 1;
	double b = -5;

	double coef3[3] = { -5, 3, 1 };
	double root1 = 2 - sqrt(5);
	double root2 = 1;
	double root3 = 2 + sqrt(5);

	double root[3];

	int nRoot = resolveRealPolynome3Laguerre(root, coef3);
	if (nRoot != 3) {
		std::cout << "wrong number of root " << nRoot << std::endl;
		return false;
	}

	bool find[3] = { false, false, false };
	for (int k = 0; k < 3; k++) {

		if (abs(root[k] - root1) < 0.001) {
			find[0] = true;
		}
		else if (abs(root[k] - root2) < 0.001) {
			find[1] = true;
		}
		else if (abs(root[k] - root3) < 0.001) {
			find[2] = true;
		}
		else {
			std::cout << "wrong root " << root[k] << " against " << root1 << " " << root2 << " " << root3 << std::endl;
			return false;
		}
	}

	for (int k = 0; k < 3; k++) {
		if (!find[k]) {
			std::cout << "wrong root " << root[0] << " " << root[1] << " " << root[2] << " against " << root1 << " " << root2 << " " << root3 << std::endl;
			return false;
		}
	}


	return true;
}
bool testresolveRealPolynome3Laguerre2() {

	double a = 1;
	double b = 0;
	double c = 3;
	double d = 1;

	double coef3[3] = { 0, 3, 1 };
	double root1 = cbrt((-1 + sqrt(5)) / 2) + cbrt((-1 - sqrt(5)) / 2);


	double root[3];

	int nRoot = resolveRealPolynome3Laguerre(root, coef3);

	if (nRoot != 1) {
		std::cout << "wrong number of root " << nRoot << std::endl;
		return false;
	}



	if (abs(root[0] - root1) > 0.001) {
		std::cout << "wrong root " << root[0] << " against " << root1 << std::endl;
		return false;
	}


	return true;
}
bool testresolveRealPolynome3Laguerre3() {
	double a = 1;
	double b = 2;
	double c = -12.75;
	double d = 11.25;

	double coef3[3] = { 2,  -12.75, 11.25 };

	double root1 = 1.5;
	double root2 = -5;



	double root[3];

	int nRoot = resolveRealPolynome3Laguerre(root, coef3);
	if (nRoot == 1) {
		std::cout << "wrong number of root " << nRoot << std::endl;
		return false;
	}

	bool find[2] = { false, false };
	for (int k = 0; k < nRoot; k++) {

		if (abs(root[k] - root1) < 0.001) {
			find[0] = true;
		}
		else if (abs(root[k] - root2) < 0.001) {
			find[1] = true;
		}
		else {
			std::cout << "wrong root " << root[k] << " against " << root1 << " " << root2 << " " << std::endl;
			return false;
		}
	}

	for (int k = 0; k < 2; k++) {
		if (!find[k]) {
			std::cout << "wrong root " << root[0] << " " << root[1] << " against " << root1 << " " << root2 << std::endl;
			return false;
		}
	}
	return true;
}


bool testresolveRealPolynome3Halley1() {

	double a = 1;
	double b = -5;

	double coef3[3] = { -5, 3, 1 };
	double root1 = 2 - sqrt(5);
	double root2 = 1;
	double root3 = 2 + sqrt(5);

	double root[3];

	int nRoot = resolveRealPolynome3Halley(root, coef3);
	if (nRoot != 3) {
		std::cout << "wrong number of root " << nRoot << std::endl;
		return false;
	}

	bool find[3] = { false, false, false };
	for (int k = 0; k < 3; k++) {

		if (abs(root[k] - root1) < 0.001) {
			find[0] = true;
		}
		else if (abs(root[k] - root2) < 0.001) {
			find[1] = true;
		}
		else if (abs(root[k] - root3) < 0.001) {
			find[2] = true;
		}
		else {
			std::cout << "wrong root " << root[k] << " against " << root1 << " " << root2 << " " << root3 << std::endl;
			return false;
		}
	}

	for (int k = 0; k < 3; k++) {
		if (!find[k]) {
			std::cout << "wrong root " << root[0] << " " << root[1] << " " << root[2] << " against " << root1 << " " << root2 << " " << root3 << std::endl;
			return false;
		}
	}


	return true;
}
bool testresolveRealPolynome3Halley2() {

	double a = 1;
	double b = 0;
	double c = 3;
	double d = 1;

	double coef3[3] = { 0, 3, 1 };
	double root1 = cbrt((-1 + sqrt(5)) / 2) + cbrt((-1 - sqrt(5)) / 2);


	double root[3];

	int nRoot = resolveRealPolynome3Halley(root, coef3);

	if (nRoot != 1) {
		std::cout << "wrong number of root " << nRoot << std::endl;
		return false;
	}



	if (abs(root[0] - root1) > 0.001) {
		std::cout << "wrong root " << root[0] << " against " << root1 << std::endl;
		return false;
	}


	return true;
}
bool testresolveRealPolynome3Halley3() {
	double a = 1;
	double b = 2;
	double c = -12.75;
	double d = 11.25;

	double coef3[3] = { 2,  -12.75, 11.25 };

	double root1 = 1.5;
	double root2 = -5;



	double root[3];

	int nRoot = resolveRealPolynome3Halley(root, coef3);
	if (nRoot == 1) {
		std::cout << "wrong number of root " << nRoot << std::endl;
		std::cout << root[0] << std::endl;
		return false;
	}

	bool find[2] = { false, false };
	for (int k = 0; k < nRoot; k++) {

		if (abs(root[k] - root1) < 0.001) {
			find[0] = true;
		}
		else if (abs(root[k] - root2) < 0.001) {
			find[1] = true;
		}
		else {
			std::cout << "wrong root " << root[k] << " against " << root1 << " " << root2 << " " << std::endl;
			return false;
		}
	}

	for (int k = 0; k < 2; k++) {
		if (!find[k]) {
			std::cout << "wrong root " << root[0] << " " << root[1] << " against " << root1 << " " << root2 << std::endl;
			return false;
		}
	}
	return true;
}

bool testresolveRealPolynome4Halley1()
{
	double rootbis[3];
	double coef4[4] = { 1, 7, 7, -6 }; //poly 3 avec tous les coefs

	double coef4bis[4] = { 6, 0, -13, 6 }; // poly 4 avec les coef sans le premier unitaire

	// juste pour trouver les racines
	double coef2[2];
	coefPolynome3From4to2coef(coef4, coef2);
	int nroot = resolveRealPolynome3without2term(rootbis, coef2);

	for (int k = 0; k < nroot; k++) {
		rootbis[k] += -coef4[1] / (3 * coef4[0]);
	}

	double root1 = rootbis[0];
	double root2 = rootbis[1];
	double root3 = rootbis[2];
	double root4 = 1;


	// vrai test
	double root[4];

	int nRoot = resolveRealPolynome4Halley(root, coef4bis);
	if (nRoot != 4) {
		std::cout << "wrong number of root " << nRoot << std::endl;
		return false;
	}

	bool find[4] = { false, false, false, false };
	for (int k = 0; k < 4; k++) { // si racine multiple ne renvoie pas d'erreur si on ne trouve pas la bonne multiplicit� mais les bonnes racines -> pas grave dans notre cas.

		if (abs(root[k] - root1) < 0.001) {
			find[0] = true;
		}
		else if (abs(root[k] - root2) < 0.001) {
			find[1] = true;
		}
		else if (abs(root[k] - root3) < 0.001) {
			find[2] = true;
		}
		else if (abs(root[k] - root4) < 0.001) {
			find[3] = true;
		}
		else {
			std::cout << "wrong root " << root[k] << " against " << root1 << " " << root2 << " " << root3 << " " << root4 << std::endl;
			return false;
		}
	}

	for (int k = 0; k < 4; k++) {
		if (!find[k]) {
			std::cout << "wrong root " << root[0] << " " << root[1] << " " << root[2] << " " << root[3] << " against " << root1 << " " << root2 << " " << root3 << " " << root4 << std::endl;
			return false;
		}
	}


	return true;
}
bool testresolveRealPolynome4Halley2()
{
	double root[4];
	double coef4[4] = { -109.778, 0, -4260.6, -3051.76 };


	int nRoot = resolveRealPolynome4Halley(root, coef4);
	if (nRoot != 2) {
		std::cout << "wrong number of root " << nRoot << std::endl;
		for (int i = 0; i < nRoot; i++) {
			std::cout << root[i] << " ";
		}
		std::cout<<std::endl;
		return false;
	}
	double rootBis[2] = { -0.707106, 110.132 };

	for (int i = 0; i < nRoot; i++) {
		double r = root[i];
		double poly = coef4[3] + coef4[2] * r + coef4[0] * r * r * r + r * r * r * r;
		if (abs(poly) > 0.000001) {
			std::cout << "wrong root poly= " << poly << " " << root[0] << " " << root[1] << " against " << rootBis[0] << " " << rootBis[1] << std::endl;
			return false;
		}
	}

	return true;
}



bool testresolveRealPolynome3GPU() {

	int nPoly = 3;
	MatrixGPUD coefs(4, nPoly);
	MatrixGPUD roots(3, nPoly, 0, 1);
	MatrixGPUD rootToFind(3, nPoly);
	MatrixGPUD nRoot(nPoly, 1, 0, 1);
	MatrixGPUD nRootToFind(nPoly, 1);

	int poly = 0;
	// --------poly 1-----------
	//double coef4[4] = { 1, -5, 3, 1 };
	coefs.set(0, poly, 1);
	coefs.set(1, poly, -5);
	coefs.set(2, poly, 3);
	coefs.set(3, poly, 1);
	// double root1 = 2 - sqrt(5); 	double root2 = 1; 	double root3 = 2 + sqrt(5);
	rootToFind.set(0, poly, 2 + sqrt(5));
	rootToFind.set(1, poly, 2 - sqrt(5));
	rootToFind.set(2, poly, 1);

	nRootToFind.set(poly, 0, 3);
	poly++;
	//--------poly 2-----------
	//double coef4[4] = { 1, 0, 3, 1 };
	coefs.set(0, poly, 1);
	coefs.set(1, poly, 0);
	coefs.set(2, poly, 3);
	coefs.set(3, poly, 1);
	// root1 = cbrt((-1 + sqrt(5)) / 2) + cbrt((-1 - sqrt(5)) / 2);
	rootToFind.set(0, poly, cbrt((-1 + sqrt(5)) / 2) + cbrt((-1 - sqrt(5)) / 2));
	nRootToFind.set(poly, 0, 1);
	poly++;


	// poly 3
	//double coef4[4] = { 1, 2,  -12.75, 11.25 };
	coefs.set(0, poly, 1);
	coefs.set(1, poly, 2);
	coefs.set(2, poly, -12.75);
	coefs.set(3, poly, 11.25);
	// double root1 = 1.5; 	double root2 = -5;
	rootToFind.set(0, poly, 1.5);
	rootToFind.set(1, poly, -5);
	nRootToFind.set(poly, 0, 2);
	poly++;

	coefs.transferGPU();

	resolveSeveralRealPolynome3GPU << <1, 32 >> > (nRoot._matrixGPU, roots._matrixGPU, coefs._matrixGPU, nPoly);

	nRoot.transferCPU();
	roots.transferCPU();

	nRoot.display();
	nRootToFind.display();

	roots.display();
	rootToFind.display();

	return true;


}
bool testresolveRealPolynome4GPU() {

	int nPoly = 2;
	MatrixGPUD coefs(4, nPoly);
	MatrixGPUD roots(4, nPoly, 0, 1);
	MatrixGPUD rootToFind(4, nPoly);
	MatrixGPUD nRoot(nPoly, 1, 0, 1);
	MatrixGPUD nRootToFind(nPoly, 1);

	int poly = 0;
	// --------poly 1-----------
	// double coef3[3] = { 6, -13, 6 };
		// determination des racines
	double rootbis[3];
	double coef4[4] = { 1, 7, 7, -6 };
	double coef2[2];
	coefPolynome3From4to2coef(coef4, coef2);
	int nroot = resolveRealPolynome3without2term(rootbis, coef2);

	coefs.set(0, poly, 6);
	coefs.set(2, poly, -13);
	coefs.set(3, poly, 6);

	for (int k = 0; k < nroot; k++) {
		rootbis[k] += -coef4[1] / (3 * coef4[0]);
	}
	rootToFind.set(0, poly, rootbis[2]);
	rootToFind.set(1, poly, rootbis[1]);
	rootToFind.set(2, poly, 1);
	rootToFind.set(3, poly, rootbis[0]);

	nRootToFind.set(poly, 0, 4);
	poly++;

	// --------poly 2-----------
// double coef3[3] = { -109.778, -4260.6, -3051.76 };

	coefs.set(0, poly, -109.778);
	coefs.set(2, poly, -4260.6);
	coefs.set(3, poly, -3051.76);
	rootToFind.set(0, poly, 110.132);
	rootToFind.set(1, poly, -0.707106);


	nRootToFind.set(poly, 0, 2);
	coefs.transferGPU();

	resolveSeveralRealPolynome4GPU << <1, 32 >> > (nRoot._matrixGPU, roots._matrixGPU, coefs._matrixGPU, nPoly);

	nRoot.transferCPU();
	roots.transferCPU();

	nRoot.display();
	nRootToFind.display();

	roots.display();
	rootToFind.display();

	return true;
}
