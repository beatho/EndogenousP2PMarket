#include "hip/hip_runtime.h"
#include "../head/System.h"
#include "../head/System.cuh"


// To DO MatrixCPU genererP0(path,dateMonth)
// To DO updateCas(P0)

System::System() {
#ifdef DEBUG_CONSTRUCTOR
	std::cout << " system constructor" << std::endl;
#endif // DEBUG_CONSTRUCTOR

	
	_simparam = Simparam(_case.getNagent(), _case.getNLine());
	_methode = nullptr;
	_result = new Simparam(_simparam);

}


System::System(float rho, int iterMaxGlobal, int iterMaxLocal, float epsGlobal, float epsLocal, std::string nameMethode, int nAgent, float P, float dP, float a, float da, float b, float db)
{
	
	int _nAgent = nAgent;
	if (nAgent == 0) {
		_case = StudyCase();
	}
	else {
		_case = StudyCase(nAgent,  P,  dP,  a,  da,  b,  db);
	}
	setMethod(nameMethode);
	

	_simparam = Simparam(rho, iterMaxGlobal, iterMaxLocal, epsGlobal, epsLocal, _nAgent);
	_result = new Simparam(_simparam);
	
}
System::~System()
{
	DELETEB(_methode);
	DELETEB(_methodePF)
	DELETEB(_methodePFGPU);
	DELETEB(_result);
}

Simparam System::solve()
{
	_methode->solve(_result, _simparam, _case);
 	return *_result;
}

Simparam System::solvePF()
{
	if(usePFGPU){
		MatrixGPU PQ = MatrixGPU(_case.getPobj(), 1);
		MatrixGPUD PQD = MatrixGPUD(_case.getPobjD(), 1);

		_methodePFGPU->init(_case, &PQ, &PQD, useDoublePF);
		_methodePFGPU->solve();
		_methodePFGPU->display(true);
	} else{
		MatrixCPU PQ = _case.getPobj();
		MatrixCPUD PQD = _case.getPobjD();

		_methodePF->init(_case, &PQ, &PQD, useDoublePF);
		_methodePF->solve();
		_methodePF->display(true);
	}


    return *_result;
}

ResultInterface* System::solve(ResultInterface* res, ParamInterface* param, StudyCaseInterface* caseInter, bool AC){
	
	
	if(AC){
		_case.SetACStudyCaseFromInterface(caseInter);
	}else{
		_case.SetDCStudyCaseFromInterface(caseInter);
	}
	_simparam.setFromInterface(param, AC);
	*_result = _simparam;
	
	_methode->solve(_result, _simparam, _case);
 	
	_result->convertToResultInterface(res);

	return res;
}
ResultInterface* System::solvePF(ResultInterface* res, ParamInterface* param, StudyCaseInterface* caseInter){
	
	MatrixCPU sizes(param->getSize());
	_case.SetACStudyCaseFromInterface(caseInter);
	if(usePFGPU){
		MatrixGPU PQ = MatrixGPU(_case.getPobj(), 1);
		MatrixGPUD PQD = MatrixGPUD(_case.getPobjD(), 1);

		_methodePFGPU->init(_case, &PQ, &PQD, useDoublePF);
		_methodePFGPU->solve();
		_methodePFGPU->display(true);
		
		res->setvarPhysic(_methodePFGPU->getW(), MatrixCPU(2*sizes.get(0, nLineP_ind), 1), _methodePFGPU->getE());
		res->setResult(_methodePFGPU->getIter(), 1, _methodePFGPU->getTime(), _methodePFGPU->getRes(), MatrixCPU(0,0));

	} else{
		MatrixCPU PQ = _case.getPobj();
		MatrixCPUD PQD = _case.getPobjD();

		_methodePF->init(_case, &PQ, &PQD, useDoublePF);
		_methodePF->solve();
		_methodePF->display(true);

		res->setvarPhysic(_methodePF->getW(), MatrixCPU(2*sizes.get(0, 2), 1), _methodePF->getE());
		res->setResult(_methodePF->getIter(), 1, _methodePF->getTime(), _methodePF->getRes(), MatrixCPU(0,0));
	}


    return res;
}


void System::solveIntervalle(std::string path, MatrixCPU* interval, int nCons, int nGen)
{
	std::ios_base::openmode mode = std::fstream::in | std::fstream::out | std::fstream::app;
	if (interval->getNCol() != 2 || interval->getNLin() != 4) {
		throw std::invalid_argument("interval must be 4*2, year, month, day, hour");
	}
	
	int Nsimu = getNbSimu(interval);

	std::cout << "Simulation count " <<Nsimu << std::endl;

	
	bool fin = false;
	bool bissextile = false;
	int year = interval->get(0, 0);
	if ((year%4 == 0 && year%100 != 0) || (year % 400 == 0)) {
		bissextile = true;
	}
	int month = interval->get(1, 0);
	int day = interval->get(2, 0);
	int hour = interval->get(3, 0);
	int dayl = dayMonth[month-1];
	if (month == 2 && bissextile) {
		dayl = 29;
	}

		
	int Nhour = 24 * dayl;
	std::string date = generateMonth(year, month);
	

	MatrixCPU P0Global(nCons, Nhour);
	
	MatrixCPU P0(nCons, 1);
	int indiceP0 = (day-1) * 24 + hour;
	
	StudyCase cas;
	_temps = MatrixCPU(1,Nsimu);
	_iter = MatrixCPU(1, Nsimu);
	_conv= MatrixCPU(1, Nsimu,-1);
	_fc = MatrixCPU(1, Nsimu);
	_ResR = MatrixCPU(1, Nsimu);
	_ResS = MatrixCPU(1, Nsimu);
	_ResX = MatrixCPU(1, Nsimu);
	int indice = 0;
	int stepG = _simparam.getStepG();
	std::string nameP0;
	
	nameP0 = path + "/load/Month/";
	generateP0(&P0Global, nameP0, date);
	P0Global.getBloc(&P0, 0, nCons, indiceP0, indiceP0 + 1);
	//P0.display();
	indiceP0 = indiceP0 + 1;
	cas.setReduce(true);
	
	cas.SetEuropeP0(path, &P0, true);
	if (_simparam._lineLimitMin) {
		cas.setLineLimitMin(_simparam._lineLimitMin); // on met une valeur minimale pour verifier si cela aide la convergence
	}
	else if (_simparam.offsetConstraint) {
		cas.setLineLimitRelaxation(_simparam.offsetConstraint);
	}
	
	//cas.display();
	 
	setStudyCase(cas);
	 
	//resetMethod();
	//display(1);
	float epsG = _simparam.getEpsG();
	clock_t t = 0;
	std::cout << "-";
	/*MatrixGPU Sensi(cas.getPowerSensi(), 1);
	MatrixGPU g(cas.getLineLimit());
	g.saveCSV("PowerLineEuropeHugeOffsetJunePowerTech.csv", mode, 1);
	g.transferGPU();*/
	while (!fin) {
		
		if (indice == (Nsimu - 1)) {
			fin = true;
		}
		
		/// simulation
		
		t = clock();
		solve();
		t = clock() - t;
		std::cout << "-";
		_temps.set(0, indice, (float)t / CLOCKS_PER_SEC);

		/*if (cas.getNLine() != 0) {
			MatrixGPU Pn(_result->getPn(), 1);
			g.multiply(&Sensi, &Pn);
			g.transferCPU();
			g.saveCSV("PowerLineEuropeOffsetJune.csv", mode, 1);
			g.transferGPU();
		}*/

		_iter.set(0, indice, _result->getIter());
		_fc.set(0, indice, _result->getFc());
		int iter = _result->getIter();
		MatrixCPU Res(_result->getRes());
		//Res.saveCSV("ResRhoRhoRho.csv", std::fstream::in | std::fstream::out | std::fstream::app);
		float resR = Res.get(0, (iter - 1) / stepG);
		float resS = Res.get(1, (iter - 1) / stepG);
		float resX = Res.get(2, (iter - 1) / stepG);
		_ResR.set(0, indice, resR);
		_ResS.set(0, indice, resS);
		_ResX.set(0, indice, resX);
		if (resR <= epsG && resS <= epsG && resX <= epsG) {
			_conv.set(0, indice, 1);
		}
		else {
			_conv.set(0, indice, 0);
		}
		if (!fin) {
			t = clock();
			/*_simparam.setTrade(&(res.getTrade()));
			_simparam.setLAMBDA(&(res.getLambda()));
			_simparam.setMU(&(res.getMU()));
			_simparam.setPn(&res.getPn());*/
			hour = hour + 1;
			if (hour == 24) {
				std::cout << day << "\n";
				day = day + 1;
				hour = 0;
				if (day > dayl) {
					day = 1;
					month = month + 1;
					if (month > 12) {
						year = year + 1;
						month = 1;
						if ((year % 4 == 0 && year % 100 != 0) || (year % 400 == 0)) {
							bissextile = true;
						}
					}
					dayl = dayMonth[month - 1];
					if (month == 2 && bissextile) {
						dayl = 29;
					}
					Nhour = 24 * dayl;
					date = generateMonth(year, month);
					P0Global.setSize(nCons, Nhour);
					generateP0(&P0Global, nameP0, date);
					indiceP0 = 0;
				}
			}
			P0Global.getBloc(&P0, 0, nCons, indiceP0, indiceP0 + 1);
			
			cas.UpdateP0(&P0);
			setStudyCase(cas);
			if (_simparam._warmstart) {
				UpdateP0();
			}
			
			indiceP0 = indiceP0 + 1;
			indice = indice + 1;
		}
		
	}
	std::cout << std::endl;
	
	std::cout << "times sum : " << _temps.sum() << std::endl;
	//std::cout << "simulation count : " << Nsimu << std::endl;
	/*std::cout << "iter :" << std::endl;
	_iter.display();
	//std::cout << "times :" << std::endl;
	//_temps.display();
	//std::cout << "conv :" << std::endl;
	//_conv.display();
	//std::cout << "fc : " << std::endl;
	//_fc.display();
	std::cout << "Res_R : " << std::endl;
	_ResR.display();
	std::cout << "Res_S : " << std::endl;
	_ResS.display();
	std::cout << "Res_X : " << std::endl;
	_ResX.display();*/
	
}


void System::solveIntervalle(std::string path, std::string name, MatrixCPU* interval)
{

	if (interval->getNCol() != 2 || interval->getNLin() != 4) {
		throw std::invalid_argument("interval must be 4*2, year, month, day, hour");
	}
	int Nsimu = getNbSimu(interval);

	std::cout << "Simulation count " << Nsimu << std::endl;


	bool fin = false;
	bool bissextile = false;
	int year = interval->get(0, 0);
	if ((year % 4 == 0 && year % 100 != 0) || (year % 400 == 0)) {
		bissextile = true;
	}
	int month = interval->get(1, 0);
	int day = interval->get(2, 0);
	int hour = interval->get(3, 0);
	int dayl = dayMonth[month - 1];
	if (month == 2 && bissextile) {
		dayl = 29;
	}


	int Nhour = 24 * dayl;
	std::string date = generateMonth(year, month);


	
	int indiceP0 = (day - 1) * 24 + hour;

	StudyCase cas;
	
	_temps = MatrixCPU(1, Nsimu);
	_iter = MatrixCPU(1, Nsimu);
	_conv = MatrixCPU(1, Nsimu, -1);
	_fc = MatrixCPU(1, Nsimu);
	_ResR = MatrixCPU(1, Nsimu);
	_ResS = MatrixCPU(1, Nsimu);
	_ResX = MatrixCPU(1, Nsimu);
	int indice = 0;
	int stepG = _simparam.getStepG();
	std::string nameP0;

	nameP0 = path + "/load/Month/";

	int nCons = getNFileline(nameP0 + date + ".txt"); // long...
		
	MatrixCPU P0Global(nCons, Nhour);
	MatrixCPU P0(nCons, 1);
	generateP0(&P0Global, nameP0, date);
	P0Global.getBloc(&P0, 0, nCons, indiceP0, indiceP0 + 1);
	//P0.display();
	indiceP0 = indiceP0 + 1;
	cas.SetStudyCase(path, name, &P0, false);
	cas.setReduce(true);
	//cas.display();

	//cas.display(2);
	setStudyCase(cas);
	try
	{
		_methode->setBestParam(cas);
	}
	catch (const std::exception&)
	{
		// juste une m�thode o� ce n'est pas implement�
	}
		
	
	//_case.display();
	//resetMethod(); grrrrrrrrrrrrrrrrrrrrrrrrr probl�me avec les matrices qui reste sur GPU
	//display(1);
	float epsG = _simparam.getEpsG();
	clock_t t = 0; 
	std::cout << "-";
	while (!fin) {

		if (indice == (Nsimu - 1)) {
			fin = true;
		}

		/// simulation

		t = clock();
		Simparam res;
		try
		{
			res = solve();
		}
		catch (const std::exception& e)
		{
			std::cout << e.what() << std::endl;
			return;
		}
		
		t = clock() - t;
		std::cout << "-";
		_temps.set(0, indice, (float)t / CLOCKS_PER_SEC);
		/*std::cout << "Echange entre les agents " << std::endl;
		//displayTradesAgent();
		std::cout << "flux dans les lignes " << std::endl;
		if (cas.getNLine() != 0) {
			MatrixCPU Sensi(cas.getPowerSensi());
			MatrixCPU Pn(res.getPn());
			MatrixCPU g(cas.getNLine(), 1, 0);
			g.multiply(&Sensi, &Pn);
			//cas.displayLineCores(&g);
		}
		std::cout << "---------------------- - " <<std::endl;*/
		//MatrixCPU lambda(res.getLambda());
		
		//lambda.display();

		_iter.set(0, indice, res.getIter());
		_fc.set(0, indice, res.getFc());
		int iter = res.getIter();
		MatrixCPU Res(res.getRes());
		//Res.saveCSV("ResRhoRhoRho.csv", std::fstream::in | std::fstream::out | std::fstream::app);
		float resR = Res.get(0, (iter - 1) / stepG);
		float resS = Res.get(1, (iter - 1) / stepG);
		float resX = Res.get(2, (iter - 1) / stepG);
		_ResR.set(0, indice, resR);
		_ResS.set(0, indice, resS);
		_ResX.set(0, indice, resX);
		if (resR <= epsG && resS <= epsG) {
			_conv.set(0, indice, 1);
		}
		else {
			_conv.set(0, indice, 0);
		}
		if (!fin) {
			t = clock();
			hour = hour + 1;
			if (hour == 24) {
				std::cout << "\n";
				day = day + 1;
				hour = 0;
				if (day > dayl) {
					day = 1;
					month = month + 1;
					if (month > 12) {
						year = year + 1;
						month = 1;
						if ((year % 4 == 0 && year % 100 != 0) || (year % 400 == 0)) {
							bissextile = true;
						}
					}
					dayl = dayMonth[month - 1];
					if (month == 2 && bissextile) {
						dayl = 29;
					}
					Nhour = 24 * dayl;
					date = generateMonth(year, month);
					P0Global.setSize(nCons, Nhour);
					generateP0(&P0Global, nameP0, date);
					indiceP0 = 0;
				}
			}
			P0Global.getBloc(&P0, 0, nCons, indiceP0, indiceP0 + 1);
			_case.UpdateP0(&P0);
			//setStudyCase(cas);
			if (_simparam._warmstart) {
				UpdateP0();
			}
			indiceP0 = indiceP0 + 1;
			indice = indice + 1;
		}

	}
	std::cout << std::endl;

	std::cout << "times sum : " << _temps.sum() << std::endl;
	//std::cout << "simulation count : " << Nsimu << std::endl;
	std::cout << "iter :" << std::endl;
	_iter.display();
	//std::cout << "times :" << std::endl;
	//_temps.display();
	//std::cout << "conv :" << std::endl;
	//_conv.display();
	//std::cout << "fc : " << std::endl;
	//_fc.display();
	/*std::cout << "Res_R : " << std::endl;
	_ResR.display();
	std::cout << "Res_S : " << std::endl;
	_ResS.display();
	std::cout << "Res_X : " << std::endl;
	_ResX.display();*/

}


void System::solveIntervalle(std::string path, int begin, int end, int chosenAgentGen)
{
	if (begin > end || begin < 0 || end > 60 * 24) {
		throw std::invalid_argument("begin and end must be withinn a day with one minute step");
	}
	int Nsimu = end - begin + 1;
	std::cout << "Simulation count " << Nsimu << std::endl;

	StudyCase cas;
	
	cas.SetEuropeTestFeeder(path, chosenAgentGen, begin);
	_simparam.setNAgentLine(cas.getNagent(), cas.getNLine(), cas.isAC());
	_result->setNAgentLine(cas.getNagent(), cas.getNLine(), cas.isAC());

	if (_methode->_name == "ADMMConst1" || _methode->_name == "ADMMGPUConst4") {
		std::cout << "ajout du DC" << std::endl;
		cas.genDCGridFromAC(); // pour utiliser les m�thodes DC
		cas.setReduce(true);
	}
	_temps = MatrixCPU(1, Nsimu);
	_iter = MatrixCPU(1, Nsimu);
	_conv = MatrixCPU(1, Nsimu, -1);
	_fc = MatrixCPU(1, Nsimu);
	_ResR = MatrixCPU(1, Nsimu);
	_ResS = MatrixCPU(1, Nsimu);
	_ResX = MatrixCPU(1, Nsimu);

	float epsG = _simparam.getEpsG();
	float epsGC = _simparam.getEpsGC();
	int stepG = _simparam.getStepG();
	clock_t t = 0;// clock();
	std::cout << "-";
	for (int iter = 0; iter < Nsimu; iter++) {
		t = clock();
		_methode->solve(_result,_simparam, cas);
		t = clock() - t;
		std::cout << "-";
		_temps.set(0, iter, (float) t / CLOCKS_PER_SEC);
		int iterFinal = _result->getIter();
		_iter.set(0, iter, iterFinal);
		_fc.set(0, iter, _result->getFc());
		MatrixCPU Res(_result->getRes());
		//Res.saveCSV("ResRhoRhoRho.csv", std::fstream::in | std::fstream::out | std::fstream::app);
		float resR = Res.get(0, (iterFinal - 1) / stepG);
		float resS = Res.get(1, (iterFinal - 1) / stepG);
		float resX = Res.get(2, (iterFinal - 1) / stepG);
		_ResR.set(0, iter, resR);
		_ResS.set(0, iter, resS);
		_ResX.set(0, iter, resX);
		if (resR <= epsG && resS <= epsG && resX <= epsGC) {
			_conv.set(0, iter, 1);
		}
		else {
			_conv.set(0, iter, 0);
		}
		if (iter < Nsimu - 1) {
			if ((iter + 1) % 60 == 0) {
				std::cout << std::endl;
			}
			cas.nextStepPobj();
			_methode->updateP0(cas);
		}
	}
	std::cout << std::endl;

	std::cout << "times sum : " << _temps.sum() << std::endl;
	//std::cout << "simulation count : " << Nsimu << std::endl;
	std::cout << "iter :" << std::endl;
	_iter.display();
	//std::cout << "times :" << std::endl;
	//_temps.display();
	//std::cout << "conv :" << std::endl;
	//_conv.display();
	//std::cout << "fc : " << std::endl;
	//_fc.display();
	/*std::cout << "Res_R : " << std::endl;
	_ResR.display();
	std::cout << "Res_S : " << std::endl;
	_ResS.display();
	std::cout << "Res_X : " << std::endl;
	_ResX.display();*/
}


void System::UpdateP0()
{
	_methode->updateP0(_case);
}

void System::resetMethod()
{
	
	setMethod(_methode->_name);
}

void System::resetParam()
{
	
	_simparam.setNAgentLine(0, 0, false);
}

void System::removeLink(int i, int j)
{
	_case.removeLink(i, j);
}

void System::addLink(int i, int j)
{
	_case.addLink(i, j);
}

Agent System::removeAgent(int agent)
{
	return _case.removeAgent(agent);
}

void System::restoreAgent(Agent& agent, bool all)
{
	_case.restoreAgent(agent, all);
}

void System::setBestRho(float rhoMax, bool rhoVar, float rhoTest)
{
	int nAgent = _case.getNagent();
	float rhoMin = 0.01;
	if (rhoMax == 0) {
		rhoMax = 0.2 * nAgent;
	}
	float epsRho = 0.01;
	float dRhoMin = 0.005;

	


	float rho_a = rhoMin;
	float rho_b = rhoMax;
	int multiplieur = 1;
	while ((rho_b - rho_a) > epsRho) {
		float dRho = multiplieur * (rho_a + rho_b) / 100;
		dRho = dRho > dRhoMin ? dRho : dRhoMin;
		float rho_x = 0.5 * (rho_a + rho_b);

		float rho_c = rho_x - (dRho / 2);
		float rho_d = rho_x + (dRho / 2);
		setRho(rho_c);
		*_result = solve();
		int iter_c = _result->getIter();
		int iterL_c = _result->getIterLTot();

		setRho(rho_d);
		*_result = solve();
		int iter_d = _result->getIter();
		int iterL_d = _result->getIterLTot();

		if (iter_d < iter_c) {
			rho_a = rho_d;
			multiplieur = 1;
		}
		else if (iter_d > iter_c) {
			rho_b = rho_c;
			multiplieur = 1;
		}
		else {
			std::cout << iter_d << " " << iter_c << " " << iterL_d << " " << iterL_c << std::endl;
			multiplieur++;
		}

	}

	float rho_x = 0.5 * (rho_a + rho_b);
	setRho(rho_x);
	*_result = solve();
	int iter_x = _result->getIter();
	int iterL_x = _result->getIterLTot();
	float time_x = _result->getTime();

	float rho_the = nAgent * 0.05;
	setRho(rho_the);
	*_result = solve();
	int iter_the = _result->getIter();
	int iterL_the = _result->getIterLTot();
	float time_the = _result->getTime();
	int iter_test = iter_the + iter_x; 
	int iterL_test = 0;
	float time_test = 0;
	if (rhoTest != 0 && rhoTest != rho_the && rhoTest != rho_x) {
		setRho(rhoTest);
		*_result = solve();
		iter_test = _result->getIter();
		iterL_test = _result->getIterLTot();
		time_test = _result->getTime();
	}

	float rhoBest = 0;
	if (iter_x < iter_the) {
		if (iter_x < iter_test) {
			rhoBest = rho_x;
		}
		else if (iter_x > iter_test) {
			rhoBest = rhoTest;
		}
		else {
			if (iterL_x <= iterL_test) {
				rhoBest = rho_x;
			}
			else {
				rhoBest = rhoTest;
			}
		}
	} else if (iter_x > iter_the) {
		if (iter_the < iter_test) {
			rhoBest = rho_the;
		}
		else if (iter_the > iter_test) {
			rhoBest = rhoTest;
		}
		else {
			if (iterL_the <= iterL_test) {
				rhoBest = rho_the;
			}
			else {
				rhoBest = rhoTest;
			}
		}
	}
	else {
		if (iter_x > iter_test) {
			rhoBest = rhoTest;	
		}
		else {
			if (iterL_the <= iterL_x) {
				rhoBest = rho_the;
			}
			else {
				rhoBest = rho_x;
			}
		}
	}
	setRho(rhoBest);
	std::cout << "Best rho find is " << rhoBest << std::endl;
	
}

void System::setStudyCase(const StudyCase& cas)
{
	_case = cas;
	if (cas.getNagent() != _simparam.getNAgent() || cas.getNLine() != _simparam.getNLine())
	{
		std::cout << "wrong number of agent or branch, simparam and result update " << std::endl;
		_simparam.setNAgentLine(cas.getNagent(), cas.getNLine(), cas.isAC());
		_result->setNAgentLine(cas.getNagent(), cas.getNLine(), cas.isAC());
		//std::cout << "end update" << std::endl;
	}

}

void System::setStudyCase(std::string fileName)
{
	_case.SetACFromFile(fileName);
	if (_case.getNagent() != _simparam.getNAgent() || _case.getNLine() != _simparam.getNLine())
	{
		std::cout << "wrong number of agent or branch, simparam and result update " << std::endl;
		_simparam.setNAgentLine(_case.getNagent(), _case.getNLine(), _case.isAC());
		_result->setNAgentLine(_case.getNagent(), _case.getNLine(), _case.isAC());
		//std::cout << "end update" << std::endl;
	}

}



void System::setSimparam(const Simparam& param)
{
	_simparam = param;
	DELETEB(_result);
	_result = new Simparam(_simparam);
	if (_case.getNagent() != param.getNAgent())
	{
		std::cout << "wrong number of agent, simparam and result update" << std::endl;
		std::cout << "if it is not wanted change the study case before doing that" << std::endl;
		_simparam.setNagent(_case.getNagent());
		_result->setNagent(_case.getNagent());
	}

}
void System::setMethod(std::string nameMethode) {
	
	DELETEB(_methode);
	useOPF = false;
	if (!nameMethode.compare(sADMMMarket)) {
		_methode = new ADMMMarket;
	}
	else if ((!nameMethode.compare(sADMMMarketMP))) {
		_methode = new ADMMMarketOpenMP;
	}else if ((!nameMethode.compare(sADMMMarketGPU))) {
		_methode = new ADMMMarketGPU;
	}
	else if (!nameMethode.compare(sADMMConst)) {
		_methode = new ADMMConst();
	}
	else if ((!nameMethode.compare(sADMMConst1))) {
		_methode = new ADMMConst1;
	}
	#ifdef OSQP
	else if ((!nameMethode.compare(sOSQPConst))) {
		_methode = new OSQPConst;
	}
	else if ((!nameMethode.compare(sADMMGPUConstCons))) {
		_methode = new ADMMGPUConstCons;
	}
	else if ((!nameMethode.compare(sDCOPFOSQP))) {
		useOPF = true;
		_methode = new DCOPFOSQP;
	}
	#endif
	else if ((!nameMethode.compare(sADMMGPUConst1))) {
		_methode = new ADMMGPUConst1;
	}
	else if ((!nameMethode.compare(sADMMGPUConst1T))) {
		_methode = new ADMMGPUConst1T;
	}
	else if ((!nameMethode.compare(sADMMGPUConst2))) {
		_methode = new ADMMGPUConst2;
	}
	else if ((!nameMethode.compare(sADMMGPUConst3))) {
		_methode = new ADMMGPUConst3;
	}
	else if ((!nameMethode.compare(sADMMGPUConst4))) {
		_methode = new ADMMGPUConst4;
	}
	else if ((!nameMethode.compare(sADMMGPUConst5))) {
		_methode = new ADMMGPUConst5;
	}
	else if ((!nameMethode.compare(sADMMGPUConstCons2))) {
		_methode = new ADMMGPUConstCons2;
	}
	else if ((!nameMethode.compare(sADMMGPUConstCons3))) {
		_methode = new ADMMGPUConstCons3;
	}
	else if ((!nameMethode.compare(sADMMACConst1))) {
		_methode = new ADMMACConst1;
	}
	else if ((!nameMethode.compare(sPAC))) {
		_methode = new PAC;
	}
	else if ((!nameMethode.compare(sPACConst))) {
		_methode = new PACConst;
	}
	else if ((!nameMethode.compare(sOPFADMM))) {
		_methode = new OPFADMM;
		useOPF = true;
	}
	else if ((!nameMethode.compare(sOPFADMMGPU))) {
		_methode = new OPFADMMGPU;
		useOPF = true;
	}else if ((!nameMethode.compare(sOPFADMM2))) {
		_methode = new OPFADMM2;
		useOPF = true;
	}
	else if ((!nameMethode.compare(sOPFADMMGPU2))) {
		_methode = new OPFADMMGPU2;
		useOPF = true;
	}
	else if ((!nameMethode.compare(sEndoMarketCons))){
		_methode = new MarEndoCons;
	}
	else if ((!nameMethode.compare(sEndoMarketConsGPU))){
		_methode = new MarEndoConsGPU;
	}
	else if ((!nameMethode.compare(sEndoMarketDirect))){
		_methode = new MarketEndoDirect;
	}
	else if ((!nameMethode.compare(sEndoMarketDirectGPU))){
		_methode = new MarketEndoDirectGPU;
	}	
	else {
		std::cout << "unknonwn method " << nameMethode << " !" << std::endl;
	}
}

void System::setMethodPF(std::string nameMethode, bool isDouble)
{
	useDoublePF = isDouble;
	if (!nameMethode.compare(sNR)) {
		_methodePF = new CPUPF;
	}
	else if ((!nameMethode.compare(sNRGPU))) {
		_methodePFGPU = new GPUPF;
		usePFGPU = true;
	}else if ((!nameMethode.compare(sGS))) {
		_methodePF = new CPUPFGS;
	}
	else if ((!nameMethode.compare(sGSGPU))) {
		_methodePFGPU = new GPUPFGS;
		usePFGPU = true;
	}
	else if ((!nameMethode.compare(sDistPQ))) {
		_methodePF = new CPUPFdistPQ;
	}
	else if ((!nameMethode.compare(sDistPQGPU))) {
		_methodePFGPU = new GPUPFdistPQ;
		usePFGPU = true;
	}else {
		std::cout << "unknonwn method " << nameMethode << " !" << std::endl;
	}
}
void System::setRho(float rho) {
	_simparam.setRho(rho);
	_result->setRho(rho);
}
void System::setRho1(float rho1)
{
	_simparam.setRho1(rho1);
	_result->setRho1(rho1);
}
void System::setRhoL(float rho)
{
	if (!useOPF) {
		((MethodP2P*) _methode)->setParam(rho);
	}
}

void System::setIter(int iterG, int iterL) {
	_simparam.setItG(iterG);
	_simparam.setItL(iterL);
	_result->setItG(iterG);
	_result->setItL(iterL);
}

void System::setItIntern(int iter)
{
	_simparam.setItIntern(iter);
}

void System::setStep(int stepG, int stepL)
{
	_simparam.setStep(stepG, stepL);
	_result->setStep(stepG, stepL);
}

void System::setStep(int stepG, int stepL, int stepIntern)
{
	_simparam.setStep(stepG, stepL, stepIntern);
}

void System::setEpsG(float epsG)
{
	_simparam.setEpsG(epsG);
}

void System::setEpsGC(float epsgC)
{
	_simparam.setEpsGC(epsgC);
}

void System::setEpsIntern(float eps)
{
	_simparam.setEpsIntern(eps);
}

void System::setEpsL(float epsL)
{
	_simparam.setEpsL(epsL);
}

void System::setTrade(MatrixCPU* trade)
{
	_simparam.setTrade(trade);
	MatrixCPU Pn(trade->getNLin(), 1);
	Pn.sum(trade);
	_simparam.setPn(&Pn);
}
void System::setLineLimitMin(float lineMin) {
	_simparam._lineLimitMin = lineMin;
}

void System::setConstraintRelaxation(float factor)
{
	float epsGC = _simparam.getEpsGC();
	_simparam.offsetConstraint = epsGC * factor;
	//_case.setLineLimitRelaxation(epsGC);
}

void System::setWarmStart(bool warmstart)
{
	_simparam._warmstart = warmstart;
}

MatrixCPU System::getRes() const {
	return _result->getRes();
}
MatrixCPU System::getTrade() const {
	return _result->getTrade();
}

MatrixCPU System::getTemps() const
{
	return _temps;
}

MatrixCPU System::getIter() const
{
	return _iter;
}

MatrixCPU System::getConv() const
{
	return _conv;
}

MatrixCPU System::getFc() const
{
	return _fc;
}

MatrixCPU System::getResR() const
{
	return _ResR;
}

MatrixCPU System::getResS() const
{
	return _ResS;
}



MatrixCPU System::getResX() const
{
	return _ResX;
}

int System::getNTrade() const
{
	return (_case.getNvoi()).sum();
}


void System::display(int type) // type=0 result, type =1 simparam & methode, type=2 case
{
	if (type==1) {
		std::cout << "Simparam : " << std::endl;
		_simparam.display(1);
		
	}
	else if (type == 2)
	{
		std::cout << "Case : " << std::endl;
		_case.display();
		_case.display(1);
		_case.display(2);
	}
	else {
		std::cout << "Method : ";
		_methode->display();
		_result->display();
	}
}

void System::displayTradesAgent()
{
	int N = _case.getNagent();
	int nCons = _case.getNCons();
	int nGen = N - nCons;
	MatrixCPU Trade(_result->getTrade());
	MatrixCPU Pn(_result->getPn());
	for (int n = 0; n < nCons; n++) {
		
		std::cout << "Agent consomateur n " << n << " Puissance echangee " << Pn.get(n,0) *_case.getSbase() << std::endl;
		for (int i = nCons; i < N; i++) {
			if (Trade.get(i, n) > 0.0001) { // positif car on regarde ce que le g�n�rateur vend
				std::cout << "         achete " << Trade.get(i, n) * _case.getSbase() << " MWh au generateur " << i << std::endl;
			}
		}
	}
}
void System::displayTime(std::string fileName) const
{
	_result->displayTime(fileName);
}

void System::setMethod(Method* method)
{
	//DELETEB(_methode); hum cela ne lui appartient pas...
	_methode = method;
}

MatrixCPU System::getPn() const
{
	return _result->getPn();
}

int System::getNFileline(std::string nameFile)
{
	int number_of_lines = 0;
	std::string line;
	std::ifstream myfile(nameFile);

	while (std::getline(myfile, line))
		++number_of_lines;
	return number_of_lines;
}

int System::getNbSimu(MatrixCPU* interval) const
{
	int year1, year2, month1, month2, day1, day2, hour1, hour2;
	year1 = interval->get(0, 0);
	year2 = interval->get(0, 1);
	month1 = interval->get(1, 0);
	month2 = interval->get(1, 1);
	day1 = interval->get(2, 0);
	day2 = interval->get(2, 1);
	hour1 = interval->get(3, 0);
	hour2 = interval->get(3, 1);
	// verifier que c'est possible
	if (year1 > year2) {
		throw std::invalid_argument("date1 must be before date2 (year)");
	}
	else if (year1 == year2) {
		if (month1 > month2) {
			throw std::invalid_argument("date1 must be before date2 (month)");
		}
		else if (month1 == month2) {
			if (day1 > day2) {
				throw std::invalid_argument("date1 must be before date2 (day)");
			}
			else if (day1 == day2) {
				if (hour1 > hour2) {
					throw std::invalid_argument("date1 must be before date2 (hour)");
				}
			}
		}
	}

	int m[12] = { 0, 31, 59, 90, 120, 151, 181, 212, 243, 273, 304, 334 };
	int dayMonth[12] = { 31, 28, 31, 30, 31, 30 , 31, 31, 30, 31, 30, 31 };

	int yearref = 2012;
	int dy = (year1 - yearref);
	int N1 = (dy * 365 + m[month1 - 1] + 1 + day1 - 1) * 24 + hour1;
	N1 = N1 + dy / 4 - dy / 100 + dy / 400;
	if ((dy % 4 == 0 && dy % 100 != 0) || (dy % 400 == 0)) {
		if (month1 < 3) {
			N1 = N1 - 24;
		}
	}

	dy = year2 - yearref;
	int N2 = (dy * 365 + m[month2 - 1] + 1 + day2 - 1) * 24 + hour2;
	N2 = N2 + dy / 4 - dy / 100 + dy / 400;

	if ((dy % 4 == 0 && dy % 100 != 0) || (dy % 400 == 0)) {
		if (month2 < 3) {
			N2 = N2 - 24;
		}
	}
	return N2 - N1 + 1;
}

int System::getNagent() const
{
	return _case.getNagent();
}



std::string System::generateDate(int year, int month, int day, int hour)
{
	std::string smonth;
	std::string sday;
	std::string shour;
	if (month < 10) {
		smonth = "0" + std::to_string(month);
	}
	else {
		smonth = std::to_string(month);
	}
	if (day < 10) {
		sday = "0" + std::to_string(day);
	}
	else {
		sday = std::to_string(day);
	}
	if (hour < 10) {
		shour = "0" + std::to_string(hour);
	}
	else {
		shour = std::to_string(hour);
	}



	std::string d = std::to_string(year) + "-" + smonth + "-" + sday + " " + shour + "-00-00";

	return d;
}
std::string System::generateMonth(int year, int month)
{
	std::string smonth;
	std::string sday;
	std::string shour;
	if (month < 10) {
		smonth = "0" + std::to_string(month);
	}
	else {
		smonth = std::to_string(month);
	}
	
	std::string d = std::to_string(year) + "-" + smonth;

	return d;
}

void System::generateP0(MatrixCPU* P0, std::string path, std::string month) {
	
	P0->setFromFile(path + month + ".txt", 1);
}
