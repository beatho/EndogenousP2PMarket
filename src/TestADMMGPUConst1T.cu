#include "hip/hip_runtime.h"
#include "../head/TestADMMGPUConst1T.cuh"
#define NSTEPLOCAL 5
#define NMAXPEERPERTRHREAD 5

int testADMMGPUConst1T()
{
	int n = 1;

	if (!testADMMGPUConst1TContruct1()) return n;
	n++;
	if (!testADMMGPUConst1TContruct2()) return n;
	n++;
	if (!testADMMGPUConst1TContruct3()) return n;
	n++;
	if (!testADMMGPUConst1TLAMBDA()) return n;
	n++;
	if (!testADMMGPUConst1TKappa()) return n;
	n++;
	if (!testADMMGPUConst1TBt1()) return n;
	n++;
	if (!testADMMGPUConst1TCP()) return n;
	n++; 
	if (!testADMMGPUConst1TCpb()) return n;
	n++;
	if (!testADMMGPUConst1TTradeP()) return n;
	n++; //10
	if (!testADMMGPUConst1TQ()) return n;
	n++; 
	if (!testADMMGPUConst1Talpha()) return n;
	n++;
	//std::cout << n << std::endl;
	if (!testADMMGPUConst1TUpdateRes()) return n;
	n++;
	if (!testADMMGPUConst1TCalcRes()) return n;
	n++;
	if (!testADMMGPUConst1TSolve1()) return n;
	n++; 
	if (!testADMMGPUConst1TSolve2()) return n;
	n++;
	if (!testADMMGPUConst1TSolve3()) return n;
	n++;
	return 0;
}

bool testADMMGPUConst1TContruct1()
{
	std::cout << "contructeur par defaut" << std::endl;
	ADMMGPUConst1T a;
	return true;
}

bool testADMMGPUConst1TContruct2()
{
	float rho = 2;

	std::cout << "contructeur avec parametres" << std::endl;
	ADMMGPUConst1T a(rho);
	return true;
}
bool testADMMGPUConst1TContruct3()
{
	float rho = 2;

	std::cout << "contructeur en deux temps" << std::endl;
	ADMMGPUConst1T a;
	a = ADMMGPUConst1T(rho);
	return true;
}

bool testADMMGPUConst1TSolve1()
{
	//solve(Simparam* result, Simparam sim, StudyCase cas);
	std::cout << "-------------------------------------------------------- " << std::endl;
	StudyCase cas;
	cas.Set2node();
	//cas.display();
	int nAgent = cas.getNagent();
	Simparam param(nAgent, 1);
	param.setRho(1);
	Simparam res(param);

	ADMMGPUConst1T a;

	a.solve(&res, param, cas);
	res.display();
	MatrixCPU Trade(nAgent, nAgent);
	Trade.set(0, 1, -1);
	Trade.set(1, 0, 1);
	MatrixCPU Res(res.getRes());
	Res.display();

	MatrixCPU trade = res.getTrade();
	trade.display();
	return trade.isEqual(&Trade, 0.001);

}
bool testADMMGPUConst1TSolve2()
{
	//solve(Simparam* result, Simparam sim, StudyCase cas);
	std::cout << "-------------------------------------------------------- " << std::endl;
	StudyCase cas;
	cas.Set29node();
	//cas.display();
	int nAgent = cas.getNagent();

	Simparam param(nAgent, cas.getNLine());
	float epsG = 0.00002f;
	float epsL = 0.000002f;
	param.setEpsL(epsL);
	param.setEpsG(epsG);
	param.setStep(1, 1);
	param.setRho(10000);
	Simparam res(param);
	ADMMGPUConst1T a;
	a.solve(&res, param, cas);
	res.display();
	MatrixCPU Trade = res.getTrade();
	MatrixCPU P2(31, 1, 0);
	P2.sum(&Trade);

	float Pn[31] = { -1.008853555,-4.62966156,-2.927534103,-0.8979898691,-0.9462603927,-0.09805059433,-0.127968356,-4.168303013,-3.151874542,-2.261414766,-0.670329392,-3.399893284,-0.4841034412,-2.775528431,-3.008597374,-1.849177122,-0.5534118414,-2.362840891,-1.122991204,-0.1379692554,-2.332088947,4.406820297,5.406073093,3.676487684,3.929354668,4.570535183,2.529039145,3.478654861,2.755935192,3.768760443,4.393183708, };

	MatrixCPU P(31, 1);
	for (int i = 0; i < 31; i++) {
		P.set(i, 0, Pn[i]);
	}
	MatrixCPU P22 = res.getPn(); 
	

	return (P2.isEqual(&P, 0.01) && P2.isEqual(&P22, 0.01));

}

bool testADMMGPUConst1TSolve3()
{
	std::cout << "-------------------------------------------------------- " << std::endl;
	StudyCase cas;
	float lim = 0.8;
	cas.Set2nodeConstraint(lim);
	int nAgent = cas.getNagent();
	Simparam param(nAgent, 1);
	Simparam res(param);
	
	param.setRho1(50);
	float value = (1 - lim) * (lim > 1) + lim;

	ADMMGPUConst1T a;

	MatrixCPU Trade(nAgent, nAgent);
	Trade.set(0, 1, -value);
	Trade.set(1, 0, value);
	a.solve(&res, param, cas);

	MatrixCPU trade = res.getTrade();
	res.display();
	trade.display();
	return trade.isEqual(&Trade, 0.001);
}

bool testADMMGPUConst1TLAMBDA()
{
	int nAgent = 3; // 2 conso et un prod
	int ntrade = 4;
	int blockSize = 256;
	int numBlocks = ceil((nAgent + blockSize - 1) / blockSize);
	float value1 = 2;
	float value2 = -8;
	float value3 = 1.5;
	float value4 = 4;
	MatrixGPU LAMBDALin(ntrade, 1, value1);
	MatrixGPU trade(ntrade, 1, value2);
	MatrixGPU CoresLinTrans(ntrade, 1);
	
	MatrixGPU LAMBDALin2(ntrade, 1, value1 + 0.5 * value3 * (value2 + value4));
	float rho = value3;


	CoresLinTrans.set(0, 0, 2);
	CoresLinTrans.set(1, 0, 3);
	CoresLinTrans.set(2, 0, 0);
	CoresLinTrans.set(3, 0, 1);

	trade.set(2, 0, value4);
	trade.set(3, 0, value4);

	
	trade.transferGPU();
	LAMBDALin.transferGPU();
	CoresLinTrans.transferGPU();
	
	updateLAMBDAGPU << <numBlocks, blockSize >> > (LAMBDALin._matrixGPU, trade._matrixGPU, rho, CoresLinTrans._matrixGPU, ntrade);

	
	LAMBDALin.transferCPU();

	return (LAMBDALin.isEqual(&LAMBDALin2));
}

bool testADMMGPUConst1TKappa()
{
	int _nLine = 3;
	
	float value1 = 2;
	float value2 = 3;
	float value3 = 1;
	float value4 = -2;
	
	int _blockSize = 256;
	int _numBlocksL = ceil((_nLine + _blockSize - 1) / _blockSize);

	MatrixCPU Qtot(_nLine, 1, value1);
	MatrixCPU Llimit(_nLine, 1, value2);
	MatrixCPU Kappa1(_nLine, 1, value3);
	MatrixCPU Kappa2(_nLine, 1, value4);
	

	Kappa1.projectNeg();
	Kappa1.add(&Llimit);
	Kappa1.subtract(&Qtot);
	Kappa2.projectNeg();
	Kappa2.add(&Llimit);
	Kappa2.add(&Qtot);

	MatrixGPU QtotGPU(_nLine, 1, value1, 1);
	MatrixGPU LlimitGPU(_nLine, 1, value2, 1);
	MatrixGPU Kappa1GPU(_nLine, 1, value3, 1);
	MatrixGPU Kappa2GPU(_nLine, 1, value4, 1);

	MatrixCPU Kappa1Result(_nLine, 1);
	MatrixCPU Kappa2Result(_nLine, 1);

	updateKappaGPU << <_numBlocksL, _blockSize >> > (Kappa1GPU._matrixGPU, Kappa2GPU._matrixGPU, LlimitGPU._matrixGPU, QtotGPU._matrixGPU, _nLine);

	Kappa1GPU.toMatCPU(Kappa1Result);
	Kappa2GPU.toMatCPU(Kappa2Result);

	

	return (Kappa1.isEqual(&Kappa1Result) && Kappa2.isEqual(&Kappa2Result));
}

bool testADMMGPUConst1TBt1()
{
	int nAgent = 3; // 2 conso et un prod
	int ntrade = 4;
	int blockSize = 256;
	int numBlocks = ceil((nAgent + blockSize - 1) / blockSize);
	float value1 = 2;
	float value2 = -8;
	float value3 = 1.5;
	float value4 = 4;
	MatrixGPU Bt1(ntrade, 1, 0);
	MatrixGPU Bt11(ntrade, 1, -value4 - value1 / value3);
	MatrixGPU trade(ntrade, 1, value2);
	MatrixGPU CoresLinTrans(ntrade, 1);

	MatrixGPU LAMBDALin(ntrade, 1, value1 + 0.5 * value3 * (value2 + value4));
	float rho = value3;


	CoresLinTrans.set(0, 0, 2);
	CoresLinTrans.set(1, 0, 3);
	CoresLinTrans.set(2, 0, 0);
	CoresLinTrans.set(3, 0, 1);

	trade.set(2, 0, value4);
	trade.set(3, 0, value4);
	Bt11.set(2, 0, -value2 - value1 / value3);
	Bt11.set(3, 0, -value2 - value1 / value3);



	Bt1.transferGPU();
	trade.transferGPU();
	LAMBDALin.transferGPU();

	CoresLinTrans.transferGPU();



	updateBt1GPU << <numBlocks, blockSize >> > (Bt1._matrixGPU, trade._matrixGPU, rho, LAMBDALin._matrixGPU, CoresLinTrans._matrixGPU, ntrade);

	Bt1.transferCPU();
	LAMBDALin.transferCPU();

	return Bt1.isEqual(&Bt11);
}

bool testADMMGPUConst1TTradeP()
{
	int nAgent = 4;
	int ntrade = 8;
	int blockSize = 256;
	//int numBlocks = ceil((nAgent + blockSize - 1) / blockSize);
	float value1 = 2;
	float value2 = 3;
	float value3 = 1;
	float value4 = 1;
	float value5 = -1;
	float value6 = -30;
	float value7 = 10;
	float value8 = 5;
	float value9 = value8 - value3 + value4 - value5;
	MatrixGPU Bt1(ntrade, 1, value1);
	float at1 = value3;
	float at2 = value4;
	MatrixGPU Ct(ntrade, 1, value5);
	MatrixGPU Lb(ntrade, 1, value6);
	MatrixGPU Ub(ntrade, 1, value7);
	MatrixGPU Tlocal(ntrade, 1);
	
	MatrixGPU Tlocal_pre(ntrade, 1, value8);
	//Tlocal_pre.set(0,0,value8+1);
	MatrixGPU Tlocal2(Tlocal_pre);
	MatrixGPU CoresLinAgent(ntrade, 1);
	CoresLinAgent.set(1, 0, 0);
	CoresLinAgent.set(2, 0, 1);
	CoresLinAgent.set(3, 0, 1);
	CoresLinAgent.set(4, 0, 2);
	CoresLinAgent.set(5, 0, 2);
	CoresLinAgent.set(6, 0, 3);
	CoresLinAgent.set(7, 0, 3);
	MatrixGPU Bp1(nAgent, 1);
	MatrixGPU Bp11(nAgent, 1);
	MatrixGPU Ap1(nAgent, 1, value2);
	MatrixGPU Ap2(nAgent, 1, value3);
	MatrixGPU Ap12(nAgent, 1, value2 + value3);
	MatrixGPU Cp(nAgent, 1, value5);
	MatrixGPU PLb(nAgent, 1, value6/2);
	MatrixGPU PUb(nAgent, 1, value7/2);

	MatrixGPU nVoisin(nAgent, 1, 2);
	MatrixGPU Tmoy(nAgent, 1, value3);
	MatrixGPU Tmoy2(Tmoy);
	MatrixGPU MU(nAgent, 1, value5);
	MatrixGPU MU2(MU);
	MatrixGPU P(nAgent, 1, value4);
	MatrixGPU P2(P);

	MatrixGPU CoresAgentLin(nAgent + 1, 1);
	CoresAgentLin.set(1, 0, 2);
	CoresAgentLin.set(2, 0, 4);
	CoresAgentLin.set(3, 0, 6);
	CoresAgentLin.set(4, 0, 8);

	
	for (int i = 0; i < nAgent; i += 1) // 1 bloc = 1 agent
	{
		float s = 0.0;
		for (int j = CoresAgentLin.get(i, 0); j < CoresAgentLin.get(i + 1, 0); j++) // on parcourt les trades de l'agent i
		{
			float m = Tlocal2.get(j, 0) - Tmoy2.get(i, 0) + P2.get(i, 0) - MU2.get(i, 0);
			float r = (Bt1.get(j, 0) * at1 + m * at2 - Ct.get(j, 0)) / (at1 + at2);
			float ub = Ub.get(j, 0);
			float lb = Lb.get(j, 0);
			float t = (ub - r) * (r > ub) + (lb - r) * (r < lb) + r;
			Tlocal2.set(j, 0, t);
			s += t;
		}
		float r = s / nVoisin.get(i, 0);
		Tmoy2.set(i, 0, r);
		Bp11.set(i, 0, r + MU2.get(i, 0));
		float p = (Ap1.get(i, 0) * Bp11.get(i, 0) - Cp.get(i, 0)) / (Ap12.get(i, 0));
		float ub = PUb.get(i, 0);
		float lb = PLb.get(i, 0);
		p = (ub - p) * (p > ub) + (lb - p) * (p < lb) + p;
		P2.set(i, 0, p);
		MU2.set(i, 0, MU2.get(i, 0) + r - P2.get(i, 0));
	}
	
	


	Bt1.transferGPU();
	Tlocal.transferGPU();
	Ct.transferGPU();
	Lb.transferGPU();
	Ub.transferGPU();
	Tlocal_pre.transferGPU();
	Tmoy.transferGPU();
	P.transferGPU();
	MU.transferGPU();
	CoresLinAgent.transferGPU();
	Ap1.transferGPU();
	Ap2.transferGPU();
	Ap12.transferGPU();
	Bp1.transferGPU();
	Cp.transferGPU();
	PLb.transferGPU();
	PUb.transferGPU();
	nVoisin.transferGPU();
	CoresAgentLin.transferGPU();

	std::cout << " fin transfert " << std::endl;


	updateTradePGPU<256> <<<nAgent, blockSize >> > (Tlocal._matrixGPU, Tlocal_pre._matrixGPU, Tmoy._matrixGPU, P._matrixGPU, MU._matrixGPU, nVoisin._matrixGPU, at1, at2, Bt1._matrixGPU, Ct._matrixGPU,
		Lb._matrixGPU, Ub._matrixGPU, Ap1._matrixGPU, Ap12._matrixGPU, Cp._matrixGPU, PLb._matrixGPU, PUb._matrixGPU, CoresAgentLin._matrixGPU, CoresLinAgent._matrixGPU, nAgent);

	Tlocal.transferCPU();
	P.transferCPU();
	MU.transferCPU();
	Tmoy.transferCPU();
	
	Tlocal2.display();
	Tlocal.display();
	Tmoy2.display();
	Tmoy.display();
	P2.display();
	P.display();
	MU2.display();
	MU.display();

	Bp11.display();


	return ((Tlocal.isEqual(&Tlocal2)) && (P.isEqual(&P2)) && (MU.isEqual(&MU2)) && (Tmoy.isEqual(&Tmoy2)));
}

bool testADMMGPUConst1Talpha()
{
	int _nAgent = 2;
	int _nLine = 3;

	float value1 = 2;
	float value2 = 3;
	
	int _blockSize = 256;
	int _numBlocksNL = ceil((_nLine*_nAgent + _blockSize - 1) / _blockSize);


	MatrixCPU Pn(_nAgent, 1, value1);
	MatrixCPU G(_nAgent, _nLine, value2);
	MatrixCPU alpha(_nAgent, _nLine);
	
	alpha.multiplyTVector(&G, &Pn, 1);
	
	MatrixGPU PnGPU(_nAgent, 1, value1, 1);
	MatrixGPU GGPU(_nAgent, _nLine, value2, 1);
	MatrixGPU alphaGPU(_nAgent, _nLine, 0, 1);

	updateAlphaTrans <<< _numBlocksNL, _blockSize >> > (alphaGPU._matrixGPU, GGPU._matrixGPU, PnGPU._matrixGPU, _nLine, _nAgent);


	MatrixCPU alphaResult(_nAgent, _nLine);
	
	alphaGPU.toMatCPU(alphaResult);
	
	std::cout << "------------------------------------ " << std::endl;
	Pn.display();
	G.display();
	alpha.display();
	

	std::cout << "------------------------------------ " << std::endl;
	alphaResult.display();


	return alpha.isEqual(&alphaResult);
}

bool testADMMGPUConst1TQ()
{
	int _nAgent = 10;
	int _nLine = 7;

	float value1 = 2;

	int _blockSize = 256;
	int _numBlocksL = ceil((_nLine + _blockSize - 1) / _blockSize);

	MatrixCPU alpha(_nAgent, _nLine, value1);
	MatrixCPU Qpart(_nAgent, _nLine);
	MatrixCPU Qtot(_nLine, 1);
	

	for (int l = 0; l < _nLine; l++) {
		float qt = 0;
		for (int n = _nAgent - 1; n >= 0; n--) {
			qt += alpha.get(n, l);
			if (n > 0) {
				Qpart.set(n - 1, l , qt);
			}
		}
		Qtot.set(l, 0, qt);
	}

	MatrixGPU alphaGPU(_nAgent, _nLine, value1, 1);
	MatrixGPU QpartGPU(_nAgent, _nLine, 0, 1);
	MatrixGPU QtotGPU(_nLine, 1, 0, 1);


	updateQpartTrans << < _nLine, _blockSize, _nAgent * sizeof(float) >> > (QpartGPU._matrixGPU, alphaGPU._matrixGPU, _nAgent, _nLine);
	updateQtotTrans << <_numBlocksL, _blockSize >> > (QtotGPU._matrixGPU, QpartGPU._matrixGPU, alphaGPU._matrixGPU, _nLine);


	MatrixCPU QpartResult(_nAgent, _nLine);
	MatrixCPU QtotResult(_nLine, 1);

	QtotGPU.toMatCPU(QtotResult);
	QpartGPU.toMatCPU(QpartResult);
	

	/*std::cout << "------------------------------------ " << std::endl;
	alpha.display();
	Qpart.display();
	Qtot.display();

	std::cout << "------------------------------------ " << std::endl;
	QpartResult.display();
	QtotResult.display();*/

	

	return (Qtot.isEqual(&QtotResult)) && (Qpart.isEqual(&QpartResult));
}

bool testADMMGPUConst1TCP()
{
	int _nAgent = 2;
	int _nLine = 3;
	float _rho1 = 1.5;
	float value1 = 2;
	float value2 = 3;
	float value3 = 1;
	float value4 = -2;
	float value5 = -1;
	float value6 = -30;
	//float value7 = 10;
	//float value8 = 5;
	int _blockSize = 256;
	int numBlocks = _nAgent;
	int _numBlocksN = ceil((_nAgent + _blockSize - 1) / _blockSize);
	int _numBlocksL = ceil((_nLine + _blockSize - 1) / _blockSize);
	
	MatrixCPU Cp(_nAgent, 1);
	MatrixCPU tempN1(_nAgent, 1);
	MatrixCPU Cp1(_nAgent, 1, value1);
	MatrixCPU Cp2(_nAgent, 1, value2);
	MatrixCPU tempL1(_nLine, 1);
	MatrixCPU Kappa1(_nLine, 1, value3);
	MatrixCPU Kappa2(_nLine, 1, value4);
	MatrixCPU G(_nAgent, _nLine, value5);
	MatrixCPU Qpart(_nAgent, _nLine, value6);
	MatrixCPU nVoisin(_nAgent, 1, 1);


	tempL1.subtractAbs(&Kappa1, &Kappa2);
	float r = 0;
	for (int i = 0; i < _nAgent; ++i)
	{
		r = 0;
		for (int k = 0; k < _nLine; ++k)
		{
			r += G.get(i, k) * (tempL1.get(k, 0) + 2 * Qpart.get(i, k));
		}
		Cp2.set(i, 0, r);
	}

	Cp2.multiply(_rho1);
	Cp2.multiplyT(&nVoisin);
	Cp.add(&Cp1, &Cp2);
	

	MatrixGPU CpGPU(_nAgent, 1, 0, 1);
	MatrixGPU tempN1GPU(_nAgent, 1, 0, 1);
	MatrixGPU Cp1GPU(_nAgent, 1, value1, 1);
	MatrixGPU Cp2GPU(_nAgent, 1, value2, 1);
	MatrixGPU tempL1GPU(_nLine, 1, 0, 1);
	MatrixGPU Kappa1GPU(_nLine, 1, value3, 1);
	MatrixGPU Kappa2GPU(_nLine, 1, value4, 1);
	MatrixGPU GGPU(_nAgent, _nLine, value5, 1);
	MatrixGPU QpartGPU(_nAgent, _nLine, value6, 1);
	MatrixCPU CpResult(_nAgent, 1);
	MatrixGPU nVoisinGPU(_nAgent, 1, 1, 1);

	diffKappa << <_numBlocksL, _blockSize >> > (tempL1GPU._matrixGPU, Kappa1GPU._matrixGPU, Kappa2GPU._matrixGPU, _nLine);
	updateCp2aTrans<256> << <numBlocks, _blockSize >> > (Cp2GPU._matrixGPU, tempL1GPU._matrixGPU, GGPU._matrixGPU, _nLine, _nAgent);
	updateCp2bTrans<256> << <numBlocks, _blockSize >> > (tempN1GPU._matrixGPU, GGPU._matrixGPU, QpartGPU._matrixGPU, _nLine, _nAgent);
	updateCpOld << <_numBlocksN, _blockSize >> > (CpGPU._matrixGPU, Cp1GPU._matrixGPU, Cp2GPU._matrixGPU, tempN1GPU._matrixGPU, nVoisinGPU._matrixGPU, _rho1, _nAgent);

	CpGPU.toMatCPU(CpResult);

	Cp.display();
	CpResult.display();

	return Cp.isEqual(&CpResult);
}

bool testADMMGPUConst1TCpb()
{
	int _nAgent = 100;
	int _nLine = 200;

	float value1 = 2;
	float value2 = 3;

	int _blockSize = 256;
	int numBlocks = _nAgent;

	
	MatrixCPU Qpart(_nAgent, _nLine, value1);
	MatrixCPU G(_nAgent, _nLine, value2);
	MatrixCPU Cpb(_nAgent, 1);

	for (int n = 0; n < _nAgent; n++) {
		float sum = 0;
		for (int l = 0; l < _nLine; l++) {
			sum += G.get(n, l) * Qpart.get(n, l);
		}
		Cpb.set(n, 0, 2*sum);
	}
	

	MatrixGPU GGPU(_nAgent, _nLine, value2, 1);
	MatrixGPU QpartGPU(_nAgent, _nLine, value1, 1);
	MatrixGPU CpbGPU(_nAgent, 1, 0, 1);

	updateCp2bTrans<256> << <numBlocks, _blockSize >> > (CpbGPU._matrixGPU, GGPU._matrixGPU, QpartGPU._matrixGPU, _nLine, _nAgent);
	
	MatrixCPU CpbResult(_nAgent, 1);
	
	CpbGPU.toMatCPU(CpbResult);
	
	return Cpb.isEqual(&CpbResult) ;

}

bool testADMMGPUConst1TUpdateRes()
{	/*float ADMMGPU5::updateRes(MatrixCPU* res, MatrixGPU* Tlocal, MatrixGPU* trade, int iter, MatrixGPU* CoresLinAgent, MatrixGPU* CoresLinVoisin, MatrixGPU* CoresMatLin, MatrixGPU* tempNN)
{
	tempNN->subtract(Tlocal, trade);
	//hipDeviceSynchronize();
	float resS = tempNN->distance2();

	updateDiffGPU <<<_numBlocks, _blockSize >>> (tempNN->_matrixGPU, Tlocal->_matrixGPU, CoresLinTrans->_matrixGPU, _N);
	//hipDeviceSynchronize();
	float resR = tempNN->distance2();

	res->set(0, iter, resR);
	res->set(1, iter, resS);


	return resR * (resR > resS) + resS * (resR <= resS);*/
	
	int ntrade = 4;
	int blockSize = 15;
	int numBlocks = ceil((ntrade + blockSize - 1) / blockSize);
	std::cout << "blockSize " << blockSize << " numBlocks " << numBlocks << std::endl;
	float value1 = 4;
	float value2 = 2.5;
	float value3 = -2;
	float value4 = value3 + value2 - value1;
	MatrixCPU res(2, 1);
	MatrixCPU res2(2, 1);
	MatrixGPU Tlocal(ntrade, 1, value1);
	MatrixGPU Tlocal_pre(ntrade, 1, value2);

	MatrixGPU CoresLinTrans(ntrade, 1);

	MatrixGPU tempN(numBlocks, 1, 0, 1);
	MatrixGPU tempN2(numBlocks, 1, 0, 1);
	
	

	CoresLinTrans.set(0, 0, 2);
	CoresLinTrans.set(1, 0, 3);
	CoresLinTrans.set(2, 0, 0);
	CoresLinTrans.set(3, 0, 1);

	Tlocal.set(2, 0, value3);
	Tlocal.set(3, 0, value3);
	Tlocal_pre.set(2, 0, value4);
	Tlocal_pre.set(3, 0, value4);



	res2.set(0, 0, sqrtf((value1 + value3) * (value1 + value3) ));
	res2.set(1, 0, sqrtf((value1 - value2) * (value1 - value2) ));
	

	Tlocal.transferGPU();
	Tlocal_pre.transferGPU();
	CoresLinTrans.transferGPU();
	
	float resS = Tlocal.max2(&Tlocal_pre);
	updateDiffGPU << <numBlocks, blockSize >> > (tempN._matrixGPU, Tlocal._matrixGPU, CoresLinTrans._matrixGPU, ntrade);
	float resR = tempN.max2();
	
	res.set(0, 0, resR);
	res.set(1, 0, resS);

	return res2.isEqual(&res);
}
bool testADMMGPUConst1TCalcRes() {
	/*
	* float ADMMGPU5::calcRes( MatrixGPU* Tlocal, MatrixGPU* P, MatrixGPU* tempN1, MatrixGPU* tempNN)
{
	 tempNN->subtract(Tlocal, &Tlocal_pre);
	 tempN1->subtract(&Tmoy, P);

	 float d1 = tempN1->max2();
	 float d2 = tempNN->max2();


	 return d1* (d1 > d2) + d2 * (d2 >= d1);
}*/

	int nAgent = 3;
	int ntrade = 4;
	int blockSize = 256;
	int numBlocks = ceil((ntrade + blockSize - 1) / blockSize);
	float value1 = 5;
	float value2 = 2;
	float value3 = -3;
	float value4 = -1;
	MatrixGPU Tlocal(ntrade, 1, value1);
	MatrixGPU Tlocal_pre(ntrade, 1, value2);
	MatrixGPU Tmoy(nAgent, 1, value3);
	MatrixGPU P(nAgent, 1, value4);


	Tlocal.transferGPU();
	Tlocal_pre.transferGPU();
	Tmoy.transferGPU();
	P.transferGPU();
	

	float d11 = Tlocal.max2(&Tlocal_pre);
	float d22 = P.max2(&Tmoy);
	float d = d11 * (d11 > d22) + d22 * (d22 >= d11);



	float d1 = fabs(value1 - value2);
	float d2 = fabs(value3 - value4);
	float df = d1 * (d1 > d2) + d2 * (d2 >= d1);



	return (df == d);
}
