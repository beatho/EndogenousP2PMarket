#include "hip/hip_runtime.h"
#include "../head/ADMMGPUConst3.cuh"

ADMMGPUConst3::ADMMGPUConst3() : MethodP2PGPU()
{
#if DEBUG_CONSTRUCTOR
	std::cout << "Constructeur ADMMGPUConst3" << std::endl;
#endif // DEBUG_CONSTRUCTOR
	_name = NAME;
	timePerBlock = MatrixCPU(1, 9, 0); // Fb0, Fb1 , Fb2, Fb3, Fb5, Fb6 Fb0'
	// si les sous ensemble ne sont pas accessible, tout est dans le premier.
	occurencePerBlock = MatrixCPU(1, 9, 0); //nb de fois utilis� pendant la simu

}


ADMMGPUConst3::ADMMGPUConst3(float rho) : MethodP2PGPU()
{
#if DEBUG_CONSTRUCTOR
	std::cout << "Constructeur ADMMGPUConst3 defaut" << std::endl;
#endif // DEBUG_CONSTRUCTOR
	_name = NAME;
	_rho = rho;
	timePerBlock = MatrixCPU(1, 9, 0); // Fb0, Fb1 , Fb2, Fb3, Fb5, Fb6 Fb0'
	// si les sous ensemble ne sont pas accessible, tout est dans le premier.
	occurencePerBlock = MatrixCPU(1, 9, 0); //nb de fois utilis� pendant la simu

}

ADMMGPUConst3::~ADMMGPUConst3()
{
}

void ADMMGPUConst3::setParam(float rho)
{
	_rho = rho;
}

void ADMMGPUConst3::setTau(float tau)
{
	if (tau < 1) {
		throw std::invalid_argument("tau must be greater than 1");
	}
	_tau = tau;
}

void ADMMGPUConst3::init(const Simparam& sim, const StudyCase& cas)
{
	// intitilisation des matrixs et variables 
	isAC = false;
	initSize(cas);
	initSimParam(sim);
	
	int nVoisinMax = nVoisin.max2();
	if (_blockSize * NMAXPEERPERTRHREAD < nVoisinMax) {
		std::cout << _blockSize << " " << NMAXPEERPERTRHREAD << " " << nVoisinMax << std::endl;
		throw std::invalid_argument("For this Method, an agent must not have more than 5120 peers");
	}
	initCaseParam(sim, cas);
	//std::cout << "mise sous forme lineaire" << std::endl;
	initLinForm(cas);

	//std::cout << "donnees sur GPU pour le grid" << std::endl;
	initDCEndoGrid(cas);
	
	//std::cout << "autres donnee sur GPU" << std::endl;
	

	initDCEndoMarket();

	
	updateGlobalProbGPU();	
}


void ADMMGPUConst3::solve(Simparam* result, const Simparam& sim, const StudyCase& cas)
{
#ifdef DEBUG_SOLVE
	cas.display();
	sim.display(1);
#endif // DEBUG_SOLVE
	
	tMarket = clock();
#ifdef INSTRUMENTATION
	std::chrono::high_resolution_clock::time_point t1;
	std::chrono::high_resolution_clock::time_point t2;
#endif // INSTRUMENTATION

	if (_id == 0) {
#ifdef INSTRUMENTATION
		hipDeviceSynchronize();
		t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION
		init(sim, cas);
#ifdef INSTRUMENTATION
		hipDeviceSynchronize();
		t2 = std::chrono::high_resolution_clock::now();
		timePerBlock.increment(0, 0, (float) std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
		occurencePerBlock.increment(0, 0, 1);
#endif // INSTRUMENTATION
	}
	//std::cout << _numBlocks2 << " " <<  _blockSize << std::endl;
	/*_mu = _muInit;
	_mu1 = _muInit;
	_rhog = sim.getRho();
	float oldRho1 = _rho1;
	_rho1 = sim.getRho1();
	float tau = _rho1 / oldRho1;
	Ap2b.multiply(_tau);
	Ap2.add(&Ap2a, &Ap2b);
	Ap12.add(&Ap1, &Ap2);*/
	_rhog = sim.getRho();
	_at1 = _rhog; // represente en fait 2*a
	
	
	float resG = 2 * _epsG;
	float resL = 2 * _epsL;
	_iterGlobal = 0;
	int iterLocal = 0;

	
	//std::cout << iterG << " " << iterL << " " << epsL << " " << epsG << std::endl;
	while ((_iterGlobal < _iterG) && (resG > _epsG)) {
		resL = 2 * _epsL;
		iterLocal = 0;
		while (iterLocal < _iterL && resL > _epsL) {
#ifdef INSTRUMENTATION
			hipDeviceSynchronize();
			t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION
			
			updateLocalProbGPU(&Tlocal, &P);
#ifdef INSTRUMENTATION
			hipDeviceSynchronize();
			t2 = std::chrono::high_resolution_clock::now();
			timePerBlock.increment(0, 1, (float) std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
#endif // INSTRUMENTATION

			if (!(iterLocal % _stepL)) {
#ifdef INSTRUMENTATION
				hipDeviceSynchronize();
				t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION
				resL = calcRes();
#ifdef INSTRUMENTATION
				hipDeviceSynchronize();
				t2 = std::chrono::high_resolution_clock::now();
				timePerBlock.increment(0, 4, (float) std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
#endif // INSTRUMENTATION
				
			}
			//std::cout << iterGlobal << " " << iterLocal << " " << resL << " " << resG << std::endl;
			Tlocal.swap(&Tlocal_pre); 
			iterLocal++;
		}
		if (iterLocal == _iterL) {
			std::cout << _iterGlobal << " " << iterLocal << " " << resL << " " << resG << std::endl;
		}
#ifdef INSTRUMENTATION
		occurencePerBlock.increment(0, 1, iterLocal);
		occurencePerBlock.increment(0, 4, iterLocal / stepL);
#endif // INSTRUMENTATION


		Tlocal.swap(&Tlocal_pre); // on �viter d'echanger lorsque qu'il ne faut pas
		tradeLin.swap(&Tlocal); // echange juste les pointeurs	
		updateGlobalProbGPU();
		if (!(_iterGlobal % _stepG)) {
#ifdef INSTRUMENTATION
			hipDeviceSynchronize();
			t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION
			resG = updateResEndo(_iterGlobal / _stepG);
#ifdef INSTRUMENTATION
			hipDeviceSynchronize();
			t2 = std::chrono::high_resolution_clock::now();
			timePerBlock.increment(0, 8, (float) std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
#endif // INSTRUMENTATION
		}
		//std::cout << iterGlobal << " " << iterLocal << " " << resL << " " << resF.get(0, iterGlobal / stepG) << " " << resF.get(1, iterGlobal / stepG) << std::endl;
		_iterGlobal++;
	}
#ifdef INSTRUMENTATION
	occurencePerBlock.increment(0, 5, iterGlobal);
	occurencePerBlock.increment(0, 6, iterGlobal);
	occurencePerBlock.increment(0, 7, iterGlobal);
	occurencePerBlock.increment(0, 8, iterGlobal / stepG);

	//std::cout << "fin simu temps : " << (float)(clock() - t) / CLOCKS_PER_SEC << std::endl;
	hipDeviceSynchronize();
	t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION


	setResult(result, cas.isAC());


#ifdef INSTRUMENTATION
	hipDeviceSynchronize();
	t2 = std::chrono::high_resolution_clock::now();
	timePerBlock.increment(0, 9, (float) std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
	occurencePerBlock.increment(0, 9, 1);
	result->setTimeBloc(&timePerBlock, &occurencePerBlock);
#endif // INSTRUMENTATION
	

	

}

void ADMMGPUConst3::updateLocalProbGPU( MatrixGPU* Tlocal, MatrixGPU* P) {
	int numBlocks = _nAgent;
	switch (_blockSize) {
	case 512:
		updateTradePGPUShared<512> << <numBlocks, _blockSize >> > (Tlocal->_matrixGPU, Tlocal_pre._matrixGPU, Tmoy._matrixGPU, P->_matrixGPU, MU._matrixGPU, nVoisin._matrixGPU, _at1, _at2, Bt1._matrixGPU, Ct._matrixGPU,
			matLb._matrixGPU, matUb._matrixGPU, Ap1._matrixGPU, Ap12._matrixGPU, Cp._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, CoresAgentLin._matrixGPU);
		break;
	case 256:
		updateTradePGPUShared<256> << <numBlocks, _blockSize >> > (Tlocal->_matrixGPU, Tlocal_pre._matrixGPU, Tmoy._matrixGPU, P->_matrixGPU, MU._matrixGPU, nVoisin._matrixGPU, _at1, _at2, Bt1._matrixGPU, Ct._matrixGPU,
			matLb._matrixGPU, matUb._matrixGPU, Ap1._matrixGPU, Ap12._matrixGPU, Cp._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, CoresAgentLin._matrixGPU);
		break;
	case 128:
		updateTradePGPUShared<128> << <numBlocks, _blockSize >> > (Tlocal->_matrixGPU, Tlocal_pre._matrixGPU, Tmoy._matrixGPU, P->_matrixGPU, MU._matrixGPU, nVoisin._matrixGPU, _at1, _at2, Bt1._matrixGPU, Ct._matrixGPU,
			matLb._matrixGPU, matUb._matrixGPU, Ap1._matrixGPU, Ap12._matrixGPU, Cp._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, CoresAgentLin._matrixGPU);
		break;
	case 64:
		updateTradePGPUShared< 64> << <numBlocks, _blockSize >> > (Tlocal->_matrixGPU, Tlocal_pre._matrixGPU, Tmoy._matrixGPU, P->_matrixGPU, MU._matrixGPU, nVoisin._matrixGPU, _at1, _at2, Bt1._matrixGPU, Ct._matrixGPU,
			matLb._matrixGPU, matUb._matrixGPU, Ap1._matrixGPU, Ap12._matrixGPU, Cp._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, CoresAgentLin._matrixGPU);
		break;
	case 32:
		updateTradePGPUShared< 32> << <numBlocks, _blockSize >> > (Tlocal->_matrixGPU, Tlocal_pre._matrixGPU, Tmoy._matrixGPU, P->_matrixGPU, MU._matrixGPU, nVoisin._matrixGPU, _at1, _at2, Bt1._matrixGPU, Ct._matrixGPU,
			matLb._matrixGPU, matUb._matrixGPU, Ap1._matrixGPU, Ap12._matrixGPU, Cp._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, CoresAgentLin._matrixGPU);
		break;
	case 16:
		updateTradePGPUShared< 16> << <numBlocks, _blockSize >> > (Tlocal->_matrixGPU, Tlocal_pre._matrixGPU, Tmoy._matrixGPU, P->_matrixGPU, MU._matrixGPU, nVoisin._matrixGPU, _at1, _at2, Bt1._matrixGPU, Ct._matrixGPU,
			matLb._matrixGPU, matUb._matrixGPU, Ap1._matrixGPU, Ap12._matrixGPU, Cp._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, CoresAgentLin._matrixGPU);
		break;
	case  8:
		updateTradePGPUShared<  8> << <numBlocks, _blockSize >> > (Tlocal->_matrixGPU, Tlocal_pre._matrixGPU, Tmoy._matrixGPU, P->_matrixGPU, MU._matrixGPU, nVoisin._matrixGPU, _at1, _at2, Bt1._matrixGPU, Ct._matrixGPU,
			matLb._matrixGPU, matUb._matrixGPU, Ap1._matrixGPU, Ap12._matrixGPU, Cp._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, CoresAgentLin._matrixGPU);
		break;
	case  4:
		updateTradePGPUShared<  4> << <numBlocks, _blockSize >> > (Tlocal->_matrixGPU, Tlocal_pre._matrixGPU, Tmoy._matrixGPU, P->_matrixGPU, MU._matrixGPU, nVoisin._matrixGPU, _at1, _at2, Bt1._matrixGPU, Ct._matrixGPU,
			matLb._matrixGPU, matUb._matrixGPU, Ap1._matrixGPU, Ap12._matrixGPU, Cp._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, CoresAgentLin._matrixGPU);
		break;
	case  2:
		updateTradePGPUShared<  2> << <numBlocks, _blockSize >> > (Tlocal->_matrixGPU, Tlocal_pre._matrixGPU, Tmoy._matrixGPU, P->_matrixGPU, MU._matrixGPU, nVoisin._matrixGPU, _at1, _at2, Bt1._matrixGPU, Ct._matrixGPU,
			matLb._matrixGPU, matUb._matrixGPU, Ap1._matrixGPU, Ap12._matrixGPU, Cp._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, CoresAgentLin._matrixGPU);
		break;
	case  1:
		updateTradePGPUShared<  1> << <numBlocks, _blockSize >> > (Tlocal->_matrixGPU, Tlocal_pre._matrixGPU, Tmoy._matrixGPU, P->_matrixGPU, MU._matrixGPU, nVoisin._matrixGPU, _at1, _at2, Bt1._matrixGPU, Ct._matrixGPU,
			matLb._matrixGPU, matUb._matrixGPU, Ap1._matrixGPU, Ap12._matrixGPU, Cp._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, CoresAgentLin._matrixGPU);
		break;
	}
}



void ADMMGPUConst3::updateGlobalProbGPU()
{
	//Rem : tout calcul qui est de taille N ou M peut �tre fait par les agents
		// Si le calcul est de taile L, soit c'est calcul� par un/des superviseurs, soit tous les agents le calcul (un peu absurde)


#ifdef INSTRUMENTATION
		// FB 3a
	hipDeviceSynchronize();
	std::chrono::high_resolution_clock::time_point t1 = std::chrono::high_resolution_clock::now();

#endif // INSTRUMENTATION

	updatePnGPU << <_numBlocksN, _blockSize >> > (Pn._matrixGPU, Tmoy._matrixGPU, nVoisin._matrixGPU, _nAgent);
	updateAlphaTrans << < _numBlocksNL, _blockSize >> > (alpha._matrixGPU, GTrans._matrixGPU, Pn._matrixGPU, _nLine, _nAgent);
	updateQpartTrans << < _nLine, _blockSize, _nAgent * sizeof(float) >> > (Qpart._matrixGPU, alpha._matrixGPU, _nAgent, _nLine);
	updateQtotTrans << <_numBlocksL, _blockSize >> > (Qtot._matrixGPU, Qpart._matrixGPU, alpha._matrixGPU, _nLine);
	
#ifdef INSTRUMENTATION
	hipDeviceSynchronize();
	std::chrono::high_resolution_clock::time_point t2 = std::chrono::high_resolution_clock::now();

	timePerBlock.increment(0, 5, (float) std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());

	// FB 3b
	hipDeviceSynchronize();
	t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION

	Kappa1_pre.set(&Kappa1);
	Kappa2_pre.set(&Kappa2);
	updateKappaGPU << <_numBlocksL, _blockSize >> > (Kappa1._matrixGPU, Kappa2._matrixGPU, lLimit._matrixGPU, Qtot._matrixGPU, _nLine);
	diffKappa << <_numBlocksL, _blockSize >> > (tempL1._matrixGPU, Kappa1._matrixGPU, Kappa2._matrixGPU, _nLine);
	int numBlocks = _nAgent;
	switch (_blockSize) {
	case 512:
		updateCp2GPUTrans<512> << <numBlocks, _blockSize >> > (Cp2._matrixGPU, tempL1._matrixGPU, GTrans._matrixGPU, Qpart._matrixGPU, nVoisin._matrixGPU, _rho1, _nLine, _nAgent);
		//updateCp2aTrans<512> << <numBlocks, _blockSize >> > (Cp2._matrixGPU, tempL1._matrixGPU, GTrans._matrixGPU, _nLine, _nAgent);
		//updateCp2bTrans<512> << <numBlocks, _blockSize >> > (tempN1._matrixGPU, GTrans._matrixGPU, Qpart._matrixGPU, _nLine, _nAgent);

		break;
	case 256:
		updateCp2GPUTrans<256> << <numBlocks, _blockSize >> > (Cp2._matrixGPU, tempL1._matrixGPU, GTrans._matrixGPU, Qpart._matrixGPU, nVoisin._matrixGPU, _rho1, _nLine, _nAgent);
		//updateCp2aTrans<256> << <numBlocks, _blockSize >> > (Cp2._matrixGPU, tempL1._matrixGPU, GTrans._matrixGPU, _nLine, _nAgent);
		//updateCp2bTrans<256> << <numBlocks, _blockSize >> > (tempN1._matrixGPU, GTrans._matrixGPU, Qpart._matrixGPU, _nLine, _nAgent);

		break;
	case 128:
		updateCp2GPUTrans<128> << <numBlocks, _blockSize >> > (Cp2._matrixGPU, tempL1._matrixGPU, GTrans._matrixGPU, Qpart._matrixGPU, nVoisin._matrixGPU, _rho1, _nLine, _nAgent);
		//updateCp2aTrans<128> << <numBlocks, _blockSize >> > (Cp2._matrixGPU, tempL1._matrixGPU, GTrans._matrixGPU, _nLine, _nAgent);
		//updateCp2bTrans<128> << <numBlocks, _blockSize >> > (tempN1._matrixGPU, GTrans._matrixGPU, Qpart._matrixGPU, _nLine, _nAgent);

		break;
	case 64:
		updateCp2GPUTrans<64> << <numBlocks, _blockSize >> > (Cp2._matrixGPU, tempL1._matrixGPU, GTrans._matrixGPU, Qpart._matrixGPU, nVoisin._matrixGPU, _rho1, _nLine, _nAgent);
		//updateCp2aTrans<64> << <numBlocks, _blockSize >> > (Cp2._matrixGPU, tempL1._matrixGPU, GTrans._matrixGPU, _nLine, _nAgent);
		//updateCp2bTrans<64> << <numBlocks, _blockSize >> > (tempN1._matrixGPU, GTrans._matrixGPU, Qpart._matrixGPU, _nLine, _nAgent);

		break;
	case 32:
		updateCp2GPUTrans<32> << <numBlocks, _blockSize >> > (Cp2._matrixGPU, tempL1._matrixGPU, GTrans._matrixGPU, Qpart._matrixGPU, nVoisin._matrixGPU, _rho1, _nLine, _nAgent);
		//updateCp2aTrans<32> << <numBlocks, _blockSize >> > (Cp2._matrixGPU, tempL1._matrixGPU, GTrans._matrixGPU, _nLine, _nAgent);
		//updateCp2bTrans<32> << <numBlocks, _blockSize >> > (tempN1._matrixGPU, GTrans._matrixGPU, Qpart._matrixGPU, _nLine, _nAgent);

		break;
	case 16:
		updateCp2GPUTrans<16> << <numBlocks, _blockSize >> > (Cp2._matrixGPU, tempL1._matrixGPU, GTrans._matrixGPU, Qpart._matrixGPU, nVoisin._matrixGPU, _rho1, _nLine, _nAgent);
		//updateCp2aTrans<16> << <numBlocks, _blockSize >> > (Cp2._matrixGPU, tempL1._matrixGPU, GTrans._matrixGPU, _nLine, _nAgent);
		//updateCp2bTrans<16> << <numBlocks, _blockSize >> > (tempN1._matrixGPU, GTrans._matrixGPU, Qpart._matrixGPU, _nLine, _nAgent);

		break;
	case  8:
		updateCp2GPUTrans<8> << <numBlocks, _blockSize >> > (Cp2._matrixGPU, tempL1._matrixGPU, GTrans._matrixGPU, Qpart._matrixGPU, nVoisin._matrixGPU, _rho1, _nLine, _nAgent);
		//updateCp2aTrans<8> << <numBlocks, _blockSize >> > (Cp2._matrixGPU, tempL1._matrixGPU, GTrans._matrixGPU, _nLine, _nAgent);
		//updateCp2bTrans<8> << <numBlocks, _blockSize >> > (tempN1._matrixGPU, GTrans._matrixGPU, Qpart._matrixGPU, _nLine, _nAgent);

		break;
	case  4:
		updateCp2GPUTrans<4> << <numBlocks, _blockSize >> > (Cp2._matrixGPU, tempL1._matrixGPU, GTrans._matrixGPU, Qpart._matrixGPU, nVoisin._matrixGPU, _rho1, _nLine, _nAgent);
		//updateCp2aTrans<4> << <numBlocks, _blockSize >> > (Cp2._matrixGPU, tempL1._matrixGPU, GTrans._matrixGPU, _nLine, _nAgent);
		//updateCp2bTrans<4> << <numBlocks, _blockSize >> > (tempN1._matrixGPU, GTrans._matrixGPU, Qpart._matrixGPU, _nLine, _nAgent);

		break;
	case  2:
		updateCp2GPUTrans<2> << <numBlocks, _blockSize >> > (Cp2._matrixGPU, tempL1._matrixGPU, GTrans._matrixGPU, Qpart._matrixGPU, nVoisin._matrixGPU, _rho1, _nLine, _nAgent);
		//updateCp2aTrans<2> << <numBlocks, _blockSize >> > (Cp2._matrixGPU, tempL1._matrixGPU, GTrans._matrixGPU, _nLine, _nAgent);
		//updateCp2bTrans<2> << <numBlocks, _blockSize >> > (tempN1._matrixGPU, GTrans._matrixGPU, Qpart._matrixGPU, _nLine, _nAgent);

		break;
	case  1:
		updateCp2GPUTrans<1> << <numBlocks, _blockSize >> > (Cp2._matrixGPU, tempL1._matrixGPU, GTrans._matrixGPU, Qpart._matrixGPU, nVoisin._matrixGPU, _rho1, _nLine, _nAgent);
		//updateCp2aTrans<1> << <numBlocks, _blockSize >> > (Cp2._matrixGPU, tempL1._matrixGPU, GTrans._matrixGPU, _nLine, _nAgent);
		//updateCp2bTrans<1> << <numBlocks, _blockSize >> > (tempN1._matrixGPU, GTrans._matrixGPU, Qpart._matrixGPU, _nLine, _nAgent);
		break;
	}

#ifdef INSTRUMENTATION
	hipDeviceSynchronize();
	t2 = std::chrono::high_resolution_clock::now();
	timePerBlock.increment(0, 6, (float) std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());

	// FB 3c
	hipDeviceSynchronize();
	t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION



	
	updateLAMBDABt1GPU << <_numBlocksM, _blockSize >> > (Bt1._matrixGPU, LAMBDALin._matrixGPU, tradeLin._matrixGPU, _rhog, CoresLinTrans._matrixGPU, _nTrade);
	updateCp << <_numBlocksN, _blockSize >> > (Cp._matrixGPU, Cp1._matrixGPU, Cp2._matrixGPU, _nAgent);

#ifdef INSTRUMENTATION
	hipDeviceSynchronize();
	t2 = std::chrono::high_resolution_clock::now();
	timePerBlock.increment(0, 7, (float) std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
#endif // INSTRUMENTATION
	

}



