#include "hip/hip_runtime.h"
#include "../head/ADMMGPUConst3.cuh"

ADMMGPUConst3::ADMMGPUConst3() : MethodP2PGPU()
{
#if DEBUG_CONSTRUCTOR
	std::cout << "Constructeur ADMMGPUConst3" << std::endl;
#endif // DEBUG_CONSTRUCTOR
	_name = NAME;
}


ADMMGPUConst3::ADMMGPUConst3(float rho) : MethodP2PGPU()
{
#if DEBUG_CONSTRUCTOR
	std::cout << "Constructeur ADMMGPUConst3 defaut" << std::endl;
#endif // DEBUG_CONSTRUCTOR
	_name = NAME;
	_rho = rho;
}

ADMMGPUConst3::~ADMMGPUConst3()
{
}

void ADMMGPUConst3::setParam(float rho)
{
	_rho = rho;
}

void ADMMGPUConst3::setTau(float tau)
{
	if (tau < 1) {
		throw std::invalid_argument("tau must be greater than 1");
	}
	_tau = tau;
}

void ADMMGPUConst3::init(const Simparam& sim, const StudyCase& cas)
{
	// intitilisation des matrixs et variables 
	

	_rhog = sim.getRho();
	_rho1 = sim.getRho1();
	//std::cout << "rho initial " << _rhog << std::endl;
	_nAgent = sim.getNAgent();
	
	_rhol = _rho;
	if (_rho == 0) {
		_rhol = _rhog;
	}
	const int iterG = sim.getIterG();
	const int stepG = sim.getStepG();
	float epsG = sim.getEpsG();
	float epsGC = sim.getEpsGC();
	_ratioEps = epsG / epsGC;
	
	nVoisinCPU = cas.getNvoi();
	nVoisin = MatrixGPU(nVoisinCPU, 1);

	int nVoisinMax = nVoisin.max2();
	if (_blockSize * NMAXPEERPERTRHREAD < nVoisinMax) {
		std::cout << _blockSize << " " << NMAXPEERPERTRHREAD << " " << nVoisinMax << std::endl;
		throw std::invalid_argument("For this Method, an agent must not have more than 5120 peers");
	}

	_nLine = cas.getNLine();
	//std::cout << _nLine << std::endl;
	_nBus = cas.getNBus();

	_nTrade = nVoisin.sum();
	_numBlocksN = ceil((_nAgent + _blockSize - 1) / _blockSize);
	_numBlocksM = ceil((_nTrade + _blockSize - 1) / _blockSize);
	_numBlocksL = ceil((_nLine + _blockSize - 1) / _blockSize);
	_numBlocksNL = ceil((_nAgent * _nLine + _blockSize - 1) / _blockSize);
	_at1 = _rhog; // represente en fait 2*a
	_at2 = _rhol;

	resF = MatrixCPU(3, (iterG / stepG) + 1);
	resX = MatrixCPU(4, (iterG / stepG) + 1);

	MatrixCPU BETA(cas.getBeta());
	MatrixGPU Ub(cas.getUb());
	MatrixGPU Lb(cas.getLb());
	LAMBDA = sim.getLambda();
	trade = sim.getTrade();
	
	//std::cout << "mise sous forme lin�aire" << std::endl;
		// Rem : si matrice d�j� existante, elles sont d�j� sur GPU donc bug pour les get
	if (Ct.getPos()) { // une copie en trop mais pour l'instant c'est ok...
		CoresMatLin.transferCPU();

		CoresLinAgent.transferCPU();
		CoresAgentLin.transferCPU();
		CoresLinVoisin.transferCPU();
		CoresLinTrans.transferCPU();

		Tlocal_pre.transferCPU();
		tradeLin.transferCPU();
		LAMBDALin.transferCPU();

		matLb.transferCPU();
		matUb.transferCPU();
		Ct.transferCPU();

	}

	CoresMatLin = MatrixGPU(_nAgent, _nAgent, -1);
	CoresLinAgent = MatrixGPU(_nTrade, 1);
	CoresAgentLin = MatrixGPU(_nAgent + 1, 1);
	CoresLinVoisin = MatrixGPU(_nTrade, 1);
	CoresLinTrans = MatrixGPU(_nTrade, 1);

	Tlocal_pre = MatrixGPU(_nTrade, 1);
	tradeLin = MatrixGPU(_nTrade, 1);
	LAMBDALin = MatrixGPU(_nTrade, 1);

	matLb = MatrixGPU(_nTrade, 1);
	matUb = MatrixGPU(_nTrade, 1);
	Ct = MatrixGPU(_nTrade, 1);

	int indice = 0;

	for (int idAgent = 0; idAgent < _nAgent; idAgent++) {
		MatrixCPU omega(cas.getVoisin(idAgent));
		int Nvoisinmax = nVoisinCPU.get(idAgent, 0);
		for (int voisin = 0; voisin < Nvoisinmax; voisin++) {
			int idVoisin = omega.get(voisin, 0);
			if(Lb.getNCol()==1){
				matLb.set(indice, 0, Lb.get(idAgent, 0));
				matUb.set(indice, 0, Ub.get(idAgent, 0));
			} else {
				matLb.set(indice, 0, Lb.get(idAgent, idVoisin));
				matUb.set(indice, 0, Ub.get(idAgent, idVoisin));
			}
			Ct.set(indice, 0, BETA.get(idAgent, idVoisin));
			tradeLin.set(indice, 0, trade.get(idAgent, idVoisin));
			Tlocal_pre.set(indice, 0, trade.get(idAgent, idVoisin));
			LAMBDALin.set(indice, 0, LAMBDA.get(idAgent, idVoisin));
			CoresLinAgent.set(indice, 0, idAgent);
			CoresLinVoisin.set(indice, 0, idVoisin);
			CoresMatLin.set(idAgent, idVoisin, indice);
			indice = indice + 1;
		}
		CoresAgentLin.set(idAgent + 1, 0, indice);
	}
	for (int lin = 0; lin < _nTrade; lin++) {
		int i = CoresLinAgent.get(lin, 0);
		int j = CoresLinVoisin.get(lin, 0);
		int k = CoresMatLin.get(j, i);
		CoresLinTrans.set(lin, 0, k);
	}


	// transfert des mises lineaire
	matUb.transferGPU();
	matLb.transferGPU();
	Ct.transferGPU();

	Tlocal_pre.transferGPU();
	tradeLin.transferGPU();
	LAMBDALin.transferGPU();

	CoresAgentLin.transferGPU();
	CoresLinAgent.transferGPU();
	CoresLinVoisin.transferGPU();
	CoresMatLin.transferGPU();
	CoresLinTrans.transferGPU();
	
	//std::cout << "donnees sur GPU pour le grid" << std::endl;
	Kappa1 = MatrixGPU(_nLine, 1, 0, 1);
	Kappa2 = MatrixGPU(_nLine, 1, 0, 1);
	Kappa1_pre = MatrixGPU(_nLine, 1, 0, 1);
	Kappa2_pre = MatrixGPU(_nLine, 1, 0, 1);
	Qpart = MatrixGPU(_nAgent, _nLine, 0, 1);
	Qtot = MatrixGPU(_nLine, 1, 0, 1);
	alpha = MatrixGPU(_nAgent, _nLine, 0, 1);

	G = MatrixGPU(cas.getPowerSensi());
	lLimit = MatrixGPU(cas.getLineLimit(), 1);
	GTrans = MatrixGPU(_nAgent, _nLine);
	
	if (GTrans.getPos()) {
		GTrans.transferCPU();
	}
	GTrans.setTrans(&G);
	//G.transferGPU();
	GTrans.transferGPU();
	G2 = GTrans;
	G2.multiplyT(&GTrans);

	//std::cout << "autres donn�e sur GPU" << std::endl;
	tempNN = MatrixGPU(_nTrade, 1, 0, 1);
	tempN1 = MatrixGPU(_nAgent, 1, 0, 1); // plut�t que de re-allouer de la m�moire � chaque utilisation
	tempL1 = MatrixGPU(_nLine, 1, 0, 1);
	tempL2 = MatrixGPU(_nLine, 1, 0, 1);
	//MatrixGPU temp1N(1, _nAgent, 0, 1);

	Tlocal = MatrixGPU(_nTrade, 1, 0, 1);
	P = MatrixGPU(_nAgent, 1, 0, 1); // moyenne des trades
	Pn = MatrixGPU(sim.getPn(), 1); // somme des trades


	a = MatrixGPU(cas.geta(), 1);
	b = MatrixGPU(cas.getb(), 1);
	Ap2 = a;
	Ap1 = nVoisin;
	Ap12 = MatrixGPU(_nAgent, 1, 0, 1);
	//Ap2a = a;
	//Ap2b = MatrixGPU(_nAgent, 1, 0, 1);

	Bt1 = MatrixGPU(_nTrade, 1, 0, 1);
	Cp = MatrixGPU(_nAgent, 1, 0, 1);
	Cp2 = MatrixGPU(_nAgent, 1, 0, 1);
	Cp1 = b;

	Pmin = MatrixGPU(cas.getPmin(), 1);
	Pmax = MatrixGPU(cas.getPmax(), 1);
	MU = MatrixGPU(sim.getMU(), 1); // facteur reduit i.e lambda_l/_rho
	Tmoy = MatrixGPU(sim.getPn(), 1);


	tempNN.preallocateReduction();
	Tlocal.preallocateReduction();
	P.preallocateReduction();
	tempL1.preallocateReduction();

	Pmin.divideT(&nVoisin);
	Pmax.divideT(&nVoisin);
	Ap1.multiply(_rhol);
	Cp1.multiplyT(&nVoisin);
	Tmoy.divideT(&nVoisin);

	tempN1.sum(&G2);
	tempN1.multiply(2 * _rho1);
	Ap2.add(&tempN1);
	Ap2.multiplyT(&nVoisin);
	Ap2.multiplyT(&nVoisin);
	Ap12.add(&Ap1, &Ap2);
	


	updateGlobalProbGPU();	
}



void ADMMGPUConst3::updateP0(const StudyCase& cas)
{
	_id = _id + 1;
#ifdef INSTRUMENTATION
	hipDeviceSynchronize();
	std::chrono::high_resolution_clock::time_point t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION

	matLb.transferCPU();
	
	Pmin = MatrixGPU(cas.getPmin());
	Pmax = MatrixGPU(cas.getPmax());


	MatrixGPU Lb(cas.getLb());

	b = cas.getb();
	Cp1 = cas.getb();
	int indice = 0;

	for (int idAgent = 0; idAgent < _nAgent; idAgent++) {
		int Nvoisinmax = nVoisinCPU.get(idAgent, 0);
		for (int voisin = 0; voisin < Nvoisinmax; voisin++) {
			matLb.set(indice, 0, Lb.get(idAgent, 0));
			indice = indice + 1;
		}
	}

	
	matLb.transferGPU();

	Pmin.divideT(&nVoisin);
	Pmax.divideT(&nVoisin);
	Cp1.multiplyT(&nVoisin);
#ifdef INSTRUMENTATION
	hipDeviceSynchronize();
	std::chrono::high_resolution_clock::time_point t2 = std::chrono::high_resolution_clock::now();
	timePerBlock.increment(0, 10, (float) std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
	occurencePerBlock.increment(0, 10, 1);
#endif // INSTRUMENTATION
	

	//std::cout << "fin update temps : " << (float)(clock() - t) / CLOCKS_PER_SEC << std::endl;
}


void ADMMGPUConst3::solve(Simparam* result, const Simparam& sim, const StudyCase& cas)
{
#ifdef DEBUG_SOLVE
	cas.display();
	sim.display(1);
#endif // DEBUG_SOLVE
	
	
	clock_t tall = clock();
#ifdef INSTRUMENTATION
	std::chrono::high_resolution_clock::time_point t1;
	std::chrono::high_resolution_clock::time_point t2;
#endif // INSTRUMENTATION

	if (_id == 0) {
#ifdef INSTRUMENTATION
		hipDeviceSynchronize();
		t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION
		init(sim, cas);
#ifdef INSTRUMENTATION
		hipDeviceSynchronize();
		t2 = std::chrono::high_resolution_clock::now();
		timePerBlock.increment(0, 0, (float) std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
		occurencePerBlock.increment(0, 0, 1);
#endif // INSTRUMENTATION
	}
	//std::cout << _numBlocks2 << " " <<  _blockSize << std::endl;
	/*_mu = _muInit;
	_mu1 = _muInit;
	_rhog = sim.getRho();
	float oldRho1 = _rho1;
	_rho1 = sim.getRho1();
	float tau = _rho1 / oldRho1;
	Ap2b.multiply(_tau);
	Ap2.add(&Ap2a, &Ap2b);
	Ap12.add(&Ap1, &Ap2);*/
	_rhog = sim.getRho();
	_at1 = _rhog; // represente en fait 2*a
	
	float epsG = sim.getEpsG();
	float epsL = sim.getEpsL();
	const int stepL = sim.getStepL();
	const int stepG = sim.getStepG();
	const int iterG = sim.getIterG();
	const int iterL = sim.getIterL();
	

	float resG = 2 * epsG;
	float resL = 2 * epsL;
	int iterGlobal = 0;
	int iterLocal = 0;

	
	//std::cout << iterG << " " << iterL << " " << epsL << " " << epsG << std::endl;
	while ((iterGlobal < iterG) && (resG>epsG)) {
		resL = 2 * epsL;
		iterLocal = 0;
		while (iterLocal< iterL && resL>epsL) {
#ifdef INSTRUMENTATION
			hipDeviceSynchronize();
			t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION
			
			updateLocalProbGPU(&Tlocal, &P);
#ifdef INSTRUMENTATION
			hipDeviceSynchronize();
			t2 = std::chrono::high_resolution_clock::now();
			timePerBlock.increment(0, 1, (float) std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
#endif // INSTRUMENTATION

			if (!(iterLocal % stepL)) {
#ifdef INSTRUMENTATION
				hipDeviceSynchronize();
				t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION
				resL = calcRes();
#ifdef INSTRUMENTATION
				hipDeviceSynchronize();
				t2 = std::chrono::high_resolution_clock::now();
				timePerBlock.increment(0, 4, (float) std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
#endif // INSTRUMENTATION
				
			}
			//std::cout << iterGlobal << " " << iterLocal << " " << resL << " " << resG << std::endl;
			Tlocal.swap(&Tlocal_pre); 
			iterLocal++;
		}
		if (iterLocal == iterL) {
			std::cout << iterGlobal << " " << iterLocal << " " << resL << " " << resG << std::endl;
		}
#ifdef INSTRUMENTATION
		occurencePerBlock.increment(0, 1, iterLocal);
		occurencePerBlock.increment(0, 4, iterLocal / stepL);
#endif // INSTRUMENTATION


		Tlocal.swap(&Tlocal_pre); // on �viter d'echanger lorsque qu'il ne faut pas
		tradeLin.swap(&Tlocal); // echange juste les pointeurs	
		updateGlobalProbGPU();
		if (!(iterGlobal % stepG)) {
#ifdef INSTRUMENTATION
			hipDeviceSynchronize();
			t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION
			resG = updateResEndo(iterGlobal / stepG);
#ifdef INSTRUMENTATION
			hipDeviceSynchronize();
			t2 = std::chrono::high_resolution_clock::now();
			timePerBlock.increment(0, 8, (float) std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
#endif // INSTRUMENTATION
		}
		//std::cout << iterGlobal << " " << iterLocal << " " << resL << " " << resF.get(0, iterGlobal / stepG) << " " << resF.get(1, iterGlobal / stepG) << std::endl;
		iterGlobal++;
	}
#ifdef INSTRUMENTATION
	occurencePerBlock.increment(0, 5, iterGlobal);
	occurencePerBlock.increment(0, 6, iterGlobal);
	occurencePerBlock.increment(0, 7, iterGlobal);
	occurencePerBlock.increment(0, 8, iterGlobal / stepG);

	//std::cout << "fin simu temps : " << (float)(clock() - t) / CLOCKS_PER_SEC << std::endl;
	hipDeviceSynchronize();
	t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION


	Kappa1.projectNeg(); //delta1
	Kappa2.projectNeg(); // delta2

	float fc = calcFc();
	//std::cout << iterGlobal << " " << iterLocal << " " << resL << " " << resG << std::endl;
	MatrixCPU tradeLinCPU;
	tradeLin.toMatCPU(tradeLinCPU);
	MatrixCPU LAMBDALinCPU;
	LAMBDALin.toMatCPU(LAMBDALinCPU);
	MatrixCPU PnCPU;
	Pn.toMatCPU(PnCPU);
	MatrixCPU MUCPU;
	MU.toMatCPU(MUCPU);
	MatrixCPU delta1CPU;
	Kappa1.toMatCPU(delta1CPU);
	MatrixCPU delta2CPU;
	Kappa2.toMatCPU(delta2CPU);
	int indice = 0;
	for (int idAgent = 0;idAgent < _nAgent; idAgent++) {
		MatrixCPU omega(cas.getVoisin(idAgent));
		int Nvoisinmax = nVoisinCPU.get(idAgent, 0);
		for (int voisin = 0; voisin < Nvoisinmax; voisin++) {
			int idVoisin = omega.get(voisin, 0);
			trade.set(idAgent, idVoisin, tradeLinCPU.get(indice, 0));
			LAMBDA.set(idAgent, idVoisin, LAMBDALinCPU.get(indice, 0));
			indice = indice + 1;
		}
	}

	result->setResF(&resF);
	result->setLAMBDA(&LAMBDA);
	result->setTrade(&trade);
	result->setDelta1(&delta1CPU);
	result->setDelta2(&delta2CPU);
	result->setIter(iterGlobal);
	result->setPn(&PnCPU);
	result->setFc(fc);
	result->setMU(&MUCPU);
	result->setRho(_rhog);


#ifdef INSTRUMENTATION
	hipDeviceSynchronize();
	t2 = std::chrono::high_resolution_clock::now();
	timePerBlock.increment(0, 9, (float) std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
	occurencePerBlock.increment(0, 9, 1);
	result->setTimeBloc(&timePerBlock, &occurencePerBlock);
#endif // INSTRUMENTATION
	

	tall = clock() - tall;
	result->setTime((float)tall / CLOCKS_PER_SEC);

}

void ADMMGPUConst3::updateLocalProbGPU( MatrixGPU* Tlocal, MatrixGPU* P) {
	int numBlocks = _nAgent;
	switch (_blockSize) {
	case 512:
		updateTradePGPUShared<512> << <numBlocks, _blockSize >> > (Tlocal->_matrixGPU, Tlocal_pre._matrixGPU, Tmoy._matrixGPU, P->_matrixGPU, MU._matrixGPU, nVoisin._matrixGPU, _at1, _at2, Bt1._matrixGPU, Ct._matrixGPU,
			matLb._matrixGPU, matUb._matrixGPU, Ap1._matrixGPU, Ap12._matrixGPU, Cp._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, CoresAgentLin._matrixGPU);
		break;
	case 256:
		updateTradePGPUShared<256> << <numBlocks, _blockSize >> > (Tlocal->_matrixGPU, Tlocal_pre._matrixGPU, Tmoy._matrixGPU, P->_matrixGPU, MU._matrixGPU, nVoisin._matrixGPU, _at1, _at2, Bt1._matrixGPU, Ct._matrixGPU,
			matLb._matrixGPU, matUb._matrixGPU, Ap1._matrixGPU, Ap12._matrixGPU, Cp._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, CoresAgentLin._matrixGPU);
		break;
	case 128:
		updateTradePGPUShared<128> << <numBlocks, _blockSize >> > (Tlocal->_matrixGPU, Tlocal_pre._matrixGPU, Tmoy._matrixGPU, P->_matrixGPU, MU._matrixGPU, nVoisin._matrixGPU, _at1, _at2, Bt1._matrixGPU, Ct._matrixGPU,
			matLb._matrixGPU, matUb._matrixGPU, Ap1._matrixGPU, Ap12._matrixGPU, Cp._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, CoresAgentLin._matrixGPU);
		break;
	case 64:
		updateTradePGPUShared< 64> << <numBlocks, _blockSize >> > (Tlocal->_matrixGPU, Tlocal_pre._matrixGPU, Tmoy._matrixGPU, P->_matrixGPU, MU._matrixGPU, nVoisin._matrixGPU, _at1, _at2, Bt1._matrixGPU, Ct._matrixGPU,
			matLb._matrixGPU, matUb._matrixGPU, Ap1._matrixGPU, Ap12._matrixGPU, Cp._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, CoresAgentLin._matrixGPU);
		break;
	case 32:
		updateTradePGPUShared< 32> << <numBlocks, _blockSize >> > (Tlocal->_matrixGPU, Tlocal_pre._matrixGPU, Tmoy._matrixGPU, P->_matrixGPU, MU._matrixGPU, nVoisin._matrixGPU, _at1, _at2, Bt1._matrixGPU, Ct._matrixGPU,
			matLb._matrixGPU, matUb._matrixGPU, Ap1._matrixGPU, Ap12._matrixGPU, Cp._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, CoresAgentLin._matrixGPU);
		break;
	case 16:
		updateTradePGPUShared< 16> << <numBlocks, _blockSize >> > (Tlocal->_matrixGPU, Tlocal_pre._matrixGPU, Tmoy._matrixGPU, P->_matrixGPU, MU._matrixGPU, nVoisin._matrixGPU, _at1, _at2, Bt1._matrixGPU, Ct._matrixGPU,
			matLb._matrixGPU, matUb._matrixGPU, Ap1._matrixGPU, Ap12._matrixGPU, Cp._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, CoresAgentLin._matrixGPU);
		break;
	case  8:
		updateTradePGPUShared<  8> << <numBlocks, _blockSize >> > (Tlocal->_matrixGPU, Tlocal_pre._matrixGPU, Tmoy._matrixGPU, P->_matrixGPU, MU._matrixGPU, nVoisin._matrixGPU, _at1, _at2, Bt1._matrixGPU, Ct._matrixGPU,
			matLb._matrixGPU, matUb._matrixGPU, Ap1._matrixGPU, Ap12._matrixGPU, Cp._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, CoresAgentLin._matrixGPU);
		break;
	case  4:
		updateTradePGPUShared<  4> << <numBlocks, _blockSize >> > (Tlocal->_matrixGPU, Tlocal_pre._matrixGPU, Tmoy._matrixGPU, P->_matrixGPU, MU._matrixGPU, nVoisin._matrixGPU, _at1, _at2, Bt1._matrixGPU, Ct._matrixGPU,
			matLb._matrixGPU, matUb._matrixGPU, Ap1._matrixGPU, Ap12._matrixGPU, Cp._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, CoresAgentLin._matrixGPU);
		break;
	case  2:
		updateTradePGPUShared<  2> << <numBlocks, _blockSize >> > (Tlocal->_matrixGPU, Tlocal_pre._matrixGPU, Tmoy._matrixGPU, P->_matrixGPU, MU._matrixGPU, nVoisin._matrixGPU, _at1, _at2, Bt1._matrixGPU, Ct._matrixGPU,
			matLb._matrixGPU, matUb._matrixGPU, Ap1._matrixGPU, Ap12._matrixGPU, Cp._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, CoresAgentLin._matrixGPU);
		break;
	case  1:
		updateTradePGPUShared<  1> << <numBlocks, _blockSize >> > (Tlocal->_matrixGPU, Tlocal_pre._matrixGPU, Tmoy._matrixGPU, P->_matrixGPU, MU._matrixGPU, nVoisin._matrixGPU, _at1, _at2, Bt1._matrixGPU, Ct._matrixGPU,
			matLb._matrixGPU, matUb._matrixGPU, Ap1._matrixGPU, Ap12._matrixGPU, Cp._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, CoresAgentLin._matrixGPU);
		break;
	}
}



void ADMMGPUConst3::updateGlobalProbGPU()
{
	//Rem : tout calcul qui est de taille N ou M peut �tre fait par les agents
		// Si le calcul est de taile L, soit c'est calcul� par un/des superviseurs, soit tous les agents le calcul (un peu absurde)


#ifdef INSTRUMENTATION
		// FB 3a
	hipDeviceSynchronize();
	std::chrono::high_resolution_clock::time_point t1 = std::chrono::high_resolution_clock::now();

#endif // INSTRUMENTATION

	updatePnGPU << <_numBlocksN, _blockSize >> > (Pn._matrixGPU, Tmoy._matrixGPU, nVoisin._matrixGPU, _nAgent);
	updateAlphaTrans << < _numBlocksNL, _blockSize >> > (alpha._matrixGPU, GTrans._matrixGPU, Pn._matrixGPU, _nLine, _nAgent);
	updateQpartTrans << < _nLine, _blockSize, _nAgent * sizeof(float) >> > (Qpart._matrixGPU, alpha._matrixGPU, _nAgent, _nLine);
	updateQtotTrans << <_numBlocksL, _blockSize >> > (Qtot._matrixGPU, Qpart._matrixGPU, alpha._matrixGPU, _nLine);
	
#ifdef INSTRUMENTATION
	hipDeviceSynchronize();
	std::chrono::high_resolution_clock::time_point t2 = std::chrono::high_resolution_clock::now();

	timePerBlock.increment(0, 5, (float) std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());

	// FB 3b
	hipDeviceSynchronize();
	t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION

	Kappa1_pre.set(&Kappa1);
	Kappa2_pre.set(&Kappa2);
	updateKappaGPU << <_numBlocksL, _blockSize >> > (Kappa1._matrixGPU, Kappa2._matrixGPU, lLimit._matrixGPU, Qtot._matrixGPU, _nLine);
	diffKappa << <_numBlocksL, _blockSize >> > (tempL1._matrixGPU, Kappa1._matrixGPU, Kappa2._matrixGPU, _nLine);
	int numBlocks = _nAgent;
	switch (_blockSize) {
	case 512:
		updateCp2GPUTrans<512> << <numBlocks, _blockSize >> > (Cp2._matrixGPU, tempL1._matrixGPU, GTrans._matrixGPU, Qpart._matrixGPU, nVoisin._matrixGPU, _rho1, _nLine, _nAgent);
		//updateCp2aTrans<512> << <numBlocks, _blockSize >> > (Cp2._matrixGPU, tempL1._matrixGPU, GTrans._matrixGPU, _nLine, _nAgent);
		//updateCp2bTrans<512> << <numBlocks, _blockSize >> > (tempN1._matrixGPU, GTrans._matrixGPU, Qpart._matrixGPU, _nLine, _nAgent);

		break;
	case 256:
		updateCp2GPUTrans<256> << <numBlocks, _blockSize >> > (Cp2._matrixGPU, tempL1._matrixGPU, GTrans._matrixGPU, Qpart._matrixGPU, nVoisin._matrixGPU, _rho1, _nLine, _nAgent);
		//updateCp2aTrans<256> << <numBlocks, _blockSize >> > (Cp2._matrixGPU, tempL1._matrixGPU, GTrans._matrixGPU, _nLine, _nAgent);
		//updateCp2bTrans<256> << <numBlocks, _blockSize >> > (tempN1._matrixGPU, GTrans._matrixGPU, Qpart._matrixGPU, _nLine, _nAgent);

		break;
	case 128:
		updateCp2GPUTrans<128> << <numBlocks, _blockSize >> > (Cp2._matrixGPU, tempL1._matrixGPU, GTrans._matrixGPU, Qpart._matrixGPU, nVoisin._matrixGPU, _rho1, _nLine, _nAgent);
		//updateCp2aTrans<128> << <numBlocks, _blockSize >> > (Cp2._matrixGPU, tempL1._matrixGPU, GTrans._matrixGPU, _nLine, _nAgent);
		//updateCp2bTrans<128> << <numBlocks, _blockSize >> > (tempN1._matrixGPU, GTrans._matrixGPU, Qpart._matrixGPU, _nLine, _nAgent);

		break;
	case 64:
		updateCp2GPUTrans<64> << <numBlocks, _blockSize >> > (Cp2._matrixGPU, tempL1._matrixGPU, GTrans._matrixGPU, Qpart._matrixGPU, nVoisin._matrixGPU, _rho1, _nLine, _nAgent);
		//updateCp2aTrans<64> << <numBlocks, _blockSize >> > (Cp2._matrixGPU, tempL1._matrixGPU, GTrans._matrixGPU, _nLine, _nAgent);
		//updateCp2bTrans<64> << <numBlocks, _blockSize >> > (tempN1._matrixGPU, GTrans._matrixGPU, Qpart._matrixGPU, _nLine, _nAgent);

		break;
	case 32:
		updateCp2GPUTrans<32> << <numBlocks, _blockSize >> > (Cp2._matrixGPU, tempL1._matrixGPU, GTrans._matrixGPU, Qpart._matrixGPU, nVoisin._matrixGPU, _rho1, _nLine, _nAgent);
		//updateCp2aTrans<32> << <numBlocks, _blockSize >> > (Cp2._matrixGPU, tempL1._matrixGPU, GTrans._matrixGPU, _nLine, _nAgent);
		//updateCp2bTrans<32> << <numBlocks, _blockSize >> > (tempN1._matrixGPU, GTrans._matrixGPU, Qpart._matrixGPU, _nLine, _nAgent);

		break;
	case 16:
		updateCp2GPUTrans<16> << <numBlocks, _blockSize >> > (Cp2._matrixGPU, tempL1._matrixGPU, GTrans._matrixGPU, Qpart._matrixGPU, nVoisin._matrixGPU, _rho1, _nLine, _nAgent);
		//updateCp2aTrans<16> << <numBlocks, _blockSize >> > (Cp2._matrixGPU, tempL1._matrixGPU, GTrans._matrixGPU, _nLine, _nAgent);
		//updateCp2bTrans<16> << <numBlocks, _blockSize >> > (tempN1._matrixGPU, GTrans._matrixGPU, Qpart._matrixGPU, _nLine, _nAgent);

		break;
	case  8:
		updateCp2GPUTrans<8> << <numBlocks, _blockSize >> > (Cp2._matrixGPU, tempL1._matrixGPU, GTrans._matrixGPU, Qpart._matrixGPU, nVoisin._matrixGPU, _rho1, _nLine, _nAgent);
		//updateCp2aTrans<8> << <numBlocks, _blockSize >> > (Cp2._matrixGPU, tempL1._matrixGPU, GTrans._matrixGPU, _nLine, _nAgent);
		//updateCp2bTrans<8> << <numBlocks, _blockSize >> > (tempN1._matrixGPU, GTrans._matrixGPU, Qpart._matrixGPU, _nLine, _nAgent);

		break;
	case  4:
		updateCp2GPUTrans<4> << <numBlocks, _blockSize >> > (Cp2._matrixGPU, tempL1._matrixGPU, GTrans._matrixGPU, Qpart._matrixGPU, nVoisin._matrixGPU, _rho1, _nLine, _nAgent);
		//updateCp2aTrans<4> << <numBlocks, _blockSize >> > (Cp2._matrixGPU, tempL1._matrixGPU, GTrans._matrixGPU, _nLine, _nAgent);
		//updateCp2bTrans<4> << <numBlocks, _blockSize >> > (tempN1._matrixGPU, GTrans._matrixGPU, Qpart._matrixGPU, _nLine, _nAgent);

		break;
	case  2:
		updateCp2GPUTrans<2> << <numBlocks, _blockSize >> > (Cp2._matrixGPU, tempL1._matrixGPU, GTrans._matrixGPU, Qpart._matrixGPU, nVoisin._matrixGPU, _rho1, _nLine, _nAgent);
		//updateCp2aTrans<2> << <numBlocks, _blockSize >> > (Cp2._matrixGPU, tempL1._matrixGPU, GTrans._matrixGPU, _nLine, _nAgent);
		//updateCp2bTrans<2> << <numBlocks, _blockSize >> > (tempN1._matrixGPU, GTrans._matrixGPU, Qpart._matrixGPU, _nLine, _nAgent);

		break;
	case  1:
		updateCp2GPUTrans<1> << <numBlocks, _blockSize >> > (Cp2._matrixGPU, tempL1._matrixGPU, GTrans._matrixGPU, Qpart._matrixGPU, nVoisin._matrixGPU, _rho1, _nLine, _nAgent);
		//updateCp2aTrans<1> << <numBlocks, _blockSize >> > (Cp2._matrixGPU, tempL1._matrixGPU, GTrans._matrixGPU, _nLine, _nAgent);
		//updateCp2bTrans<1> << <numBlocks, _blockSize >> > (tempN1._matrixGPU, GTrans._matrixGPU, Qpart._matrixGPU, _nLine, _nAgent);
		break;
	}

#ifdef INSTRUMENTATION
	hipDeviceSynchronize();
	t2 = std::chrono::high_resolution_clock::now();
	timePerBlock.increment(0, 6, (float) std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());

	// FB 3c
	hipDeviceSynchronize();
	t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION



	
	updateLAMBDABt1GPU << <_numBlocksM, _blockSize >> > (Bt1._matrixGPU, LAMBDALin._matrixGPU, tradeLin._matrixGPU, _rhog, CoresLinTrans._matrixGPU, _nTrade);
	updateCp << <_numBlocksN, _blockSize >> > (Cp._matrixGPU, Cp1._matrixGPU, Cp2._matrixGPU, _nAgent);

#ifdef INSTRUMENTATION
	hipDeviceSynchronize();
	t2 = std::chrono::high_resolution_clock::now();
	timePerBlock.increment(0, 7, (float) std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
#endif // INSTRUMENTATION
	

}
void ADMMGPUConst3::display() {

	std::cout << _name << std::endl;
}


