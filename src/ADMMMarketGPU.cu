#include "hip/hip_runtime.h"
#include "../head/ADMMMarketGPU.cuh"
#define MAX(X, Y) X * (X >= Y) + Y * (Y > X)


ADMMMarketGPU::ADMMMarketGPU() : MethodP2P()
{
#if DEBUG_CONSTRUCTOR
	std::cout << " ADMMMarketGPU Constructor" << std::endl;
#endif // DEBUG_CONSTRUCTOR
	_name = NAME;
}


ADMMMarketGPU::ADMMMarketGPU(float rho) : MethodP2P()
{
#if DEBUG_CONSTRUCTOR
	std::cout << "default ADMMMarketGPU Constructor" << std::endl;
#endif // DEBUG_CONSTRUCTOR
	_name = NAME;
	_rho = rho;
}

ADMMMarketGPU::~ADMMMarketGPU()
{
}
void ADMMMarketGPU::setParam(float rho)
{
	_rho = rho;
}

void ADMMMarketGPU::setTau(float tau)
{
	if (tau < 1) {
		throw std::invalid_argument("tau must be greater than 1");
	}
	_tau = tau;
}

void ADMMMarketGPU::solveWithMinPower(Simparam* result, const Simparam& sim, const StudyCase& cas)
{
	init(sim, cas);
	int nCons = cas.getNCons();


	for (int n = 0; n < nCons; n++) {
		Pmax.set(n, 0, Pmin.get(n, 0));
	}

	solve(result, sim, cas);
}


void ADMMMarketGPU::solve(Simparam* result, const Simparam& sim, const StudyCase& cas)
{
#ifdef DEBUG_SOLVE
	cas.display();
	sim.display(1);
#endif // DEBUG_SOLVE
	
	tMarket =clock();
#ifdef INSTRUMENTATION
	std::chrono::high_resolution_clock::time_point t1;
	std::chrono::high_resolution_clock::time_point t2;
#endif // INSTRUMENTATION

	// FB 0
	if (_id == 0) {
#ifdef INSTRUMENTATION
		hipDeviceSynchronize();
		t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION
		init(sim, cas);
#ifdef INSTRUMENTATION
		hipDeviceSynchronize();
		t2 = std::chrono::high_resolution_clock::now();
		timePerBlock.increment(0, 0, std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
		occurencePerBlock.increment(0, 0, 1);
#endif // INSTRUMENTATION
	}
	_rhog = sim.getRho();
	_at1 = _rhog;

	int iterL = sim.getIterL();
	int stepL = sim.getStepL();
	float epsL = sim.getEpsL() / 5;
	float epsG = sim.getEpsG();
	
	float epsL2 = epsL * epsL;
	float fc = 0;

	int iterLocal = 0;
	float resG = 2 * epsG;
	float resL = 2 * epsL;
	_iterGlobal = 0;

	while (((_iterGlobal < _iterG) && (resG>epsG)) ) {
		/*P.saveCSVForce("testPGPU2.csv", 11, 1);
		Tlocal.saveCSVForce("testTGPU2.csv", 11, 1);
		Bt1.saveCSVForce("testBGPU2.csv", 11, 1);*/

		//std::cout << "lambda" << std::endl;
		//LAMBDALin.display(true);
#ifdef INSTRUMENTATION
		hipDeviceSynchronize();
		t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION
		updateLocalProbGPU(epsL2, iterL);
#ifdef INSTRUMENTATION
		hipDeviceSynchronize();
		t2 = std::chrono::high_resolution_clock::now();
		timePerBlock.increment(0, 1, std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
#endif // INSTRUMENTATION
		//Tlocal_pre.swap(&Tlocal);
		tradeLin.swap(&Tlocal);
		
#ifdef INSTRUMENTATION
		hipDeviceSynchronize();
		t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION
		updateGlobalProb();
#ifdef INSTRUMENTATION
		hipDeviceSynchronize();
		t2 = std::chrono::high_resolution_clock::now();
		timePerBlock.increment(0, 5, std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
#endif // INSTRUMENTATION
		// FB 4
		if (!(_iterGlobal % _stepG)) {
#ifdef INSTRUMENTATION
			t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION
			resG = updateRes(&resF, (_iterGlobal / _stepG), &tempNN);
#ifdef INSTRUMENTATION
			t2 = std::chrono::high_resolution_clock::now();
			timePerBlock.increment(0, 6, std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
#endif // INSTRUMENTATION
		}
		//std::cout << iterGlobal << " " << iterLocal << " " << resL << " " << resF.get(0, iterGlobal / stepG) << " " << resF.get(1, iterGlobal / stepG) << std::endl;

		_iterGlobal++;
	}
	//std::cout << _iterGlobal << " " << iterLocal << " " << resL << " " << resF.get(0, (_iterGlobal - 1) / _stepG) << " " << resF.get(1, (_iterGlobal - 1) / _stepG) << " " << resF.get(2, (_iterGlobal - 1) / _stepG) << std::endl;
#ifdef INSTRUMENTATION	

	hipDeviceSynchronize();
	occurencePerBlock.increment(0, 1, _iterGlobal);
	occurencePerBlock.increment(0, 5, _iterGlobal);
	occurencePerBlock.increment(0, 6, _iterGlobal / _stepG);
	t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION
	
	
	updatePn(&Pn, &P, &nVoisin);

	/*std::cout << "power" << std::endl;
	Tmoy.display(true);
	P.display(true);
	Pn.display(true); 

	std::cout << "lambda" << std::endl;
	LAMBDALin.display(true);*/

	fc = calcFc(&a, &b, &tradeLin, &Pn, &Ct, &tempN1, &tempNN);
	MatrixCPU tradeLinCPU;
	tradeLin.toMatCPU(tradeLinCPU);
	MatrixCPU LAMBDALinCPU;
	LAMBDALin.toMatCPU(LAMBDALinCPU);
	MatrixCPU PnCPU;
	Pn.toMatCPU(PnCPU);
	MatrixCPU MUCPU;
	MU.toMatCPU(MUCPU);
	int indice = 0;
	for (int idAgent = 0; idAgent < _nAgentTrue; idAgent++) {
		MatrixCPU omega(cas.getVoisin(idAgent));
		int Nvoisinmax = nVoisinCPU.get(idAgent, 0);
		for (int voisin = 0; voisin < Nvoisinmax; voisin++) {
			int idVoisin = omega.get(voisin, 0);
			trade.set(idAgent, idVoisin, tradeLinCPU.get(indice, 0));
			LAMBDA.set(idAgent, idVoisin, LAMBDALinCPU.get(indice, 0));
			indice = indice + 1;
		}
	}
	for (int idAgent = _nAgentTrue; idAgent < _nAgent; idAgent++) {
		for (int idVoisin = 0; idVoisin < _nAgentTrue; idVoisin++) {
			if (idVoisin != (idAgent - _nAgentTrue)) {
				trade.set(idAgent, idVoisin, tradeLinCPU.get(indice, 0));
				LAMBDA.set(idAgent, idVoisin, LAMBDALinCPU.get(indice, 0));
				indice = indice + 1;
			}

		}
	}

	result->setResF(&resF);
	result->setLAMBDA(&LAMBDA);
	result->setTrade(&trade);
	result->setIter(_iterGlobal);
	result->setPn(&PnCPU);
	result->setFc(fc);
	result->setMU(&MUCPU);
	result->setRho(_rhog);

#ifdef INSTRUMENTATION
	hipDeviceSynchronize();
	t2 = std::chrono::high_resolution_clock::now();
	timePerBlock.increment(0, 7, std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
	occurencePerBlock.increment(0, 7, 1);
	result->setTimeBloc(&timePerBlock, &occurencePerBlock);
#endif // INSTRUMENTATION

	tMarket = clock() - tMarket;

	result->setTime((float)tMarket / CLOCKS_PER_SEC);
}

void ADMMMarketGPU::updateP0(const StudyCase& cas)
{
	_id = _id + 1;
#ifdef INSTRUMENTATION
	std::chrono::high_resolution_clock::time_point t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION

	matLb.transferCPU();
	Pmin = MatrixGPU(cas.getPmin());
	Pmax = MatrixGPU(cas.getPmax());


	MatrixCPU Lb(cas.getLb());

	b = cas.getb();
	Cp = b;
	int indice = 0;

	for (int idAgent = 0; idAgent < _nAgentTrue; idAgent++) {
		int Nvoisinmax = nVoisinCPU.get(idAgent, 0);
		for (int voisin = 0; voisin < Nvoisinmax; voisin++) {
			matLb.set(indice, 0, Lb.get(idAgent, 0));
			indice = indice + 1;
		}
	}
	for (int idAgent = _nAgentTrue; idAgent < _nAgent; idAgent++) {
		for (int voisin = 0; voisin < (_nAgent - 1); voisin++) {
			matLb.set(indice, 0, Lb.get(idAgent, 0));
			indice = indice + 1;
		}
	}

	matLb.transferGPU();

	Pmin.divideT(&nVoisin);
	Pmax.divideT(&nVoisin);
	Cp.multiplyT(&nVoisin);
#ifdef INSTRUMENTATION
	std::chrono::high_resolution_clock::time_point t2 = std::chrono::high_resolution_clock::now();
	timePerBlock.increment(0, 8, std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
	occurencePerBlock.increment(0, 8, 1);
#endif // INSTRUMENTATION

}

void ADMMMarketGPU::init(const Simparam& sim, const StudyCase& cas)
{
	// intitilisation des matrixs et variables 
	
	clock_t t = clock();

	isAC = cas.isAC();
	//std::cout << "init " << std::endl;
	_rhog = sim.getRho();
	
	_iterG = sim.getIterG();
	_stepG = sim.getStepG();
	float epsG = sim.getEpsG();
	
	_nAgentTrue = sim.getNAgent();
	_nAgent = _nAgentTrue + isAC * _nAgentTrue;

	_rhol = _rho; //*nAgent
	//std::cout << "rho " << _rho << std::endl;
	if (_rho == 0) {
		_rhol = _rhog;
	}

	nVoisinCPU = cas.getNvoi();
	nVoisin = MatrixGPU(nVoisinCPU, 1);
	nVoisin.preallocateReduction();

	_nTrade = nVoisin.sum();
	_nTradeP = 0;
	if (isAC) {
		for (int n = 0; n < _nAgentTrue; n++) {
			_nTradeP += nVoisinCPU.get(n, 0);
		}
		_nTradeQ = _nTrade - _nTradeP;
		if (_nTradeQ != (_nAgentTrue * (_nAgentTrue - 1))) {
			std::cout << "err ADMMGPU : " << _nAgent << " " << _nAgentTrue << " " << _nTrade << " " << _nTradeP << " " << _nTradeQ << std::endl;

			throw std::invalid_argument("Agent must be fully conected for the Q echanges, WIP");
		}
	}
	else {
		_nTradeP = _nTrade;
	}
	//std::cout << isAC << " " <<  _nAgentTrue << " " << _nAgent << " " << _nTrade << " " << _nTradeP << " " << _nTradeQ << std::endl;
	_numBlocksN = ceil((_nAgent + _blockSize - 1) / _blockSize);
	_numBlocksM = ceil((_nTrade + _blockSize - 1) / _blockSize);
	

	_at1 = _rhog; 
	_at2 = _rhol;

	resF = MatrixCPU(3, (_iterG / _stepG) + 1);

	MatrixCPU BETA(cas.getBeta());
	MatrixCPU Ub(cas.getUb());
	MatrixCPU Lb(cas.getLb());
	LAMBDA = sim.getLambda();
	trade = sim.getTrade();

	//std::cout << "mise sous forme lin�aire" << std::endl;
	if (Ct.getPos()) { // une copie en trop mais pour l'instant c'est ok...
		CoresMatLin.transferCPU();

		CoresLinAgent.transferCPU();
		CoresAgentLin.transferCPU();
		CoresLinVoisin.transferCPU();
		CoresLinTrans.transferCPU();

		Tlocal_pre.transferCPU();
		tradeLin.transferCPU();
		LAMBDALin.transferCPU();

		matLb.transferCPU();
		matUb.transferCPU();
		Ct.transferCPU();
	}
	
	CoresMatLin = MatrixGPU(_nAgent, _nAgentTrue, -1);
	CoresAgentLin = MatrixGPU( _nAgent + 1, 1);
	CoresLinAgent = MatrixGPU(_nTrade, 1);
	CoresLinVoisin = MatrixGPU(_nTrade, 1);
	CoresLinTrans = MatrixGPU(_nTrade, 1);

	Tlocal_pre = MatrixGPU(_nTrade, 1);
	tradeLin = MatrixGPU(_nTrade, 1);
	LAMBDALin = MatrixGPU(_nTrade, 1);

	matLb = MatrixGPU(_nTrade, 1);
	matUb = MatrixGPU(_nTrade, 1);
	Ct = MatrixGPU(_nTrade, 1);
	

	int indice = 0;
	//std::cout << " P " << std::endl;
	for (int idAgent = 0; idAgent < _nAgentTrue; idAgent++) { // P
		MatrixCPU omega(cas.getVoisin(idAgent));
		int Nvoisinmax = nVoisinCPU.get(idAgent, 0);
		for (int voisin = 0; voisin < Nvoisinmax; voisin++) {
			int idVoisin = omega.get(voisin, 0);
			matLb.set(indice, 0, Lb.get(idAgent, 0));
			matUb.set(indice, 0, Ub.get(idAgent, 0));
			Ct.set(indice, 0, BETA.get(idAgent, idVoisin));
			tradeLin.set(indice, 0, trade.get(idAgent, idVoisin));
			Tlocal_pre.set(indice, 0, trade.get(idAgent, idVoisin));
			LAMBDALin.set(indice, 0, LAMBDA.get(idAgent, idVoisin));
			CoresLinAgent.set(indice, 0, idAgent);
			CoresLinVoisin.set(indice, 0, idVoisin);
			CoresMatLin.set(idAgent, idVoisin, indice);
			indice = indice + 1;
		}
		CoresAgentLin.set(idAgent + 1, 0, indice);
	}
	//std::cout << " Q " << std::endl;
	for (int idAgent = _nAgentTrue; idAgent < _nAgent; idAgent++) { // Q
		for (int idVoisin = 0; idVoisin < _nAgentTrue; idVoisin++) {
			if (idVoisin != (idAgent - _nAgentTrue)) {
				matLb.set(indice, 0, Lb.get(idAgent, 0));
				matUb.set(indice, 0, Ub.get(idAgent, 0));
				//Ct.set(indice, 0, BETA.get(idAgent, idVoisin));
				tradeLin.set(indice, 0, trade.get(idAgent, idVoisin));
				Tlocal_pre.set(indice, 0, trade.get(idAgent, idVoisin));
				LAMBDALin.set(indice, 0, LAMBDA.get(idAgent, idVoisin));
				CoresLinAgent.set(indice, 0, idAgent );
				CoresLinVoisin.set(indice, 0, idVoisin+_nAgentTrue);
				CoresMatLin.set(idAgent, idVoisin, indice);
				indice = indice + 1;
			}
		}
		CoresAgentLin.set(idAgent + 1, 0, indice);
	}
	for (int lin = 0; lin < _nTrade; lin++) {
		int i = CoresLinAgent.get(lin, 0);
		int j = CoresLinVoisin.get(lin, 0);
		if (lin >= _nTradeP) {
			i -= _nAgentTrue;
		}
				
		int k = CoresMatLin.get(j, i);
		CoresLinTrans.set(lin, 0, k);
	}

	// transfert des mises lineaire
	matUb.transferGPU();
	matLb.transferGPU();
	Ct.transferGPU();

	Tlocal_pre.transferGPU();
	tradeLin.transferGPU();
	LAMBDALin.transferGPU();

	CoresAgentLin.transferGPU();
	CoresLinAgent.transferGPU();
	CoresLinVoisin.transferGPU();
	CoresMatLin.transferGPU();
	CoresLinTrans.transferGPU();
	/*std::cout << "trade bound" << std::endl;
	matLb.display();
    matUb.display();*/

	

	//std::cout << "autres donn�e sur GPU" << std::endl;
	tempNN = MatrixGPU(_nTrade, 1, 0, 1);
	tempN1 = MatrixGPU(_nAgent, 1, 0, 1); // plut�t que de re-allouer de la m�moire � chaque utilisation
	//MatrixCPU temp1N(1, _nAgent, 0, 1);

	Tlocal = MatrixGPU(_nTrade, 1, 0, 1);
	
	
	Pn = MatrixGPU(sim.getPn(), 1); // somme des trades
	P = Pn;// moyenne des trades, ici c'est juste pour qu'il ait la m�me taille sans avoir besoin de se poser de question
	
	a = MatrixGPU(cas.geta(), 1);
	b = MatrixGPU(cas.getb(), 1);

	Ap2 = a;
	Ap1 = nVoisin;
	Ap12 = MatrixGPU(_nAgent, 1, 0, 1);

	Bt1 = MatrixGPU(_nTrade, 1, 0, 1);
	Cp = b;
	

	Pmin = MatrixGPU(cas.getPmin(), 1);
	Pmax = MatrixGPU(cas.getPmax(), 1);
	MU = MatrixGPU(sim.getMU(), 1); // facteur reduit i.e lambda_l/_rho
	Tmoy = MatrixGPU(sim.getPn(), 1);

	tempNN.preallocateReduction();
	Tlocal.preallocateReduction();
	P.preallocateReduction();

	Pmin.divideT(&nVoisin);
	Pmax.divideT(&nVoisin);

	/*std::cout << "Power bound" << std::endl;
	Pmin.display();
	Pmax.display();*/

	Ap1.multiply(_rhol);
	Cp.multiplyT(&nVoisin);
	Tmoy.divideT(&nVoisin);
	
	Ap2.multiplyT(&nVoisin);
	Ap2.multiplyT(&nVoisin);
	Ap12.add(&Ap1, &Ap2);


	updateGlobalProb();
	//std::cout << "rho " << _rhog << " rhoL " << _rhol << " rho1 " << _rho1 << std::endl;
	//std::cout << "fin init temps : " << (float)(clock() - t) / CLOCKS_PER_SEC << std::endl;
	

}

void ADMMMarketGPU::updateGlobalProb() {
	updateLAMBDABt1GPU <<<_numBlocksM, _blockSize >> > (Bt1._matrixGPU, LAMBDALin._matrixGPU, tradeLin._matrixGPU, _rhog, CoresLinTrans._matrixGPU, _nTrade);
}

void ADMMMarketGPU::updateLocalProbGPU(float epsL, int nIterL)
{
	int numBlocks = _nAgent;
	switch (_blockSize) {
	case 512:
		updateTradePGPUSharedResidual<512> << <numBlocks, _blockSize >> > (Tlocal._matrixGPU, Tlocal_pre._matrixGPU, Tmoy._matrixGPU, P._matrixGPU, MU._matrixGPU, nVoisin._matrixGPU, _at1, _at2, Bt1._matrixGPU, Ct._matrixGPU,
			matLb._matrixGPU, matUb._matrixGPU, Ap1._matrixGPU, Ap12._matrixGPU, Cp._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, CoresAgentLin._matrixGPU, epsL, nIterL);
		break;
	case 256:
		updateTradePGPUSharedResidual<256> << <numBlocks, _blockSize >> > (Tlocal._matrixGPU, Tlocal_pre._matrixGPU, Tmoy._matrixGPU, P._matrixGPU, MU._matrixGPU, nVoisin._matrixGPU, _at1, _at2, Bt1._matrixGPU, Ct._matrixGPU,
			matLb._matrixGPU, matUb._matrixGPU, Ap1._matrixGPU, Ap12._matrixGPU, Cp._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, CoresAgentLin._matrixGPU, epsL, nIterL);
		break;
	case 128:
		updateTradePGPUSharedResidual<128> << <numBlocks, _blockSize >> > (Tlocal._matrixGPU, Tlocal_pre._matrixGPU, Tmoy._matrixGPU, P._matrixGPU, MU._matrixGPU, nVoisin._matrixGPU, _at1, _at2, Bt1._matrixGPU, Ct._matrixGPU,
			matLb._matrixGPU, matUb._matrixGPU, Ap1._matrixGPU, Ap12._matrixGPU, Cp._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, CoresAgentLin._matrixGPU, epsL, nIterL);
		break;
	case 64:
		updateTradePGPUSharedResidual< 64> << <numBlocks, _blockSize >> > (Tlocal._matrixGPU, Tlocal_pre._matrixGPU, Tmoy._matrixGPU, P._matrixGPU, MU._matrixGPU, nVoisin._matrixGPU, _at1, _at2, Bt1._matrixGPU, Ct._matrixGPU,
			matLb._matrixGPU, matUb._matrixGPU, Ap1._matrixGPU, Ap12._matrixGPU, Cp._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, CoresAgentLin._matrixGPU, epsL, nIterL);
		break;
	case 32:
		updateTradePGPUSharedResidual< 32> << <numBlocks, _blockSize >> > (Tlocal._matrixGPU, Tlocal_pre._matrixGPU, Tmoy._matrixGPU, P._matrixGPU, MU._matrixGPU, nVoisin._matrixGPU, _at1, _at2, Bt1._matrixGPU, Ct._matrixGPU,
			matLb._matrixGPU, matUb._matrixGPU, Ap1._matrixGPU, Ap12._matrixGPU, Cp._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, CoresAgentLin._matrixGPU, epsL, nIterL);
		break;
	case 16:
		updateTradePGPUSharedResidual< 16> << <numBlocks, _blockSize >> > (Tlocal._matrixGPU, Tlocal_pre._matrixGPU, Tmoy._matrixGPU, P._matrixGPU, MU._matrixGPU, nVoisin._matrixGPU, _at1, _at2, Bt1._matrixGPU, Ct._matrixGPU,
			matLb._matrixGPU, matUb._matrixGPU, Ap1._matrixGPU, Ap12._matrixGPU, Cp._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, CoresAgentLin._matrixGPU, epsL, nIterL);
		break;
	case  8:
		updateTradePGPUSharedResidual<  8> << <numBlocks, _blockSize >> > (Tlocal._matrixGPU, Tlocal_pre._matrixGPU, Tmoy._matrixGPU, P._matrixGPU, MU._matrixGPU, nVoisin._matrixGPU, _at1, _at2, Bt1._matrixGPU, Ct._matrixGPU,
			matLb._matrixGPU, matUb._matrixGPU, Ap1._matrixGPU, Ap12._matrixGPU, Cp._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, CoresAgentLin._matrixGPU, epsL, nIterL);
		break;
	case  4:
		updateTradePGPUSharedResidual<  4> << <numBlocks, _blockSize >> > (Tlocal._matrixGPU, Tlocal_pre._matrixGPU, Tmoy._matrixGPU, P._matrixGPU, MU._matrixGPU, nVoisin._matrixGPU, _at1, _at2, Bt1._matrixGPU, Ct._matrixGPU,
			matLb._matrixGPU, matUb._matrixGPU, Ap1._matrixGPU, Ap12._matrixGPU, Cp._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, CoresAgentLin._matrixGPU, epsL, nIterL);
		break;
	case  2:
		updateTradePGPUSharedResidual<  2> << <numBlocks, _blockSize >> > (Tlocal._matrixGPU, Tlocal_pre._matrixGPU, Tmoy._matrixGPU, P._matrixGPU, MU._matrixGPU, nVoisin._matrixGPU, _at1, _at2, Bt1._matrixGPU, Ct._matrixGPU,
			matLb._matrixGPU, matUb._matrixGPU, Ap1._matrixGPU, Ap12._matrixGPU, Cp._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, CoresAgentLin._matrixGPU, epsL, nIterL);
		break;
	case  1:
		updateTradePGPUSharedResidual<  1> << <numBlocks, _blockSize >> > (Tlocal._matrixGPU, Tlocal_pre._matrixGPU, Tmoy._matrixGPU, P._matrixGPU, MU._matrixGPU, nVoisin._matrixGPU, _at1, _at2, Bt1._matrixGPU, Ct._matrixGPU,
			matLb._matrixGPU, matUb._matrixGPU, Ap1._matrixGPU, Ap12._matrixGPU, Cp._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, CoresAgentLin._matrixGPU, epsL, nIterL);
		break;
	}
}



float ADMMMarketGPU::updateRes(MatrixCPU* res, int iter, MatrixGPU* tempNN)
{
	//std::cout << "tradeLin" << std::endl;
	//tradeLin.display(true);
	// 
	updateDiffGPU << <_numBlocksM, _blockSize >> > (tempNN->_matrixGPU, tradeLin._matrixGPU, CoresLinTrans._matrixGPU, _nTrade);
	
	
	//std::cout << "tempNN" << std::endl;
	//tempNN->display(true);
	
	float resR = tempNN->max2();

	float resS = Tlocal.max2(&tradeLin); // nomalement * _rhog mais si _rhog est tres grand impossible que cela converge !!!
	
	//std::cout << iter << " " << resR << " " << resS << std::endl;
	if (iter > 0) {
		if (resR > _mu * resS) {
			_rhog = _tau * _rhog;
			_at1 = _rhog;
			//std::cout << iter << ", rho augmente :" << _rhog << std::endl;
		}
		else if (resS > _mu * resR) {// rho = rho / tau_inc;
			_rhog = _rhog / _tau;
			_at1 = _rhog;
			//std::cout << iter << ", rho diminue :" << _rhog << std::endl;
		}
	}/**/
	
	
	res->set(0, iter, resR);
	res->set(1, iter, resS);
	
	return MAX(resS, resR);
}


void ADMMMarketGPU::display() {

	if (_iterGlobal == 0) {
		std::cout << "algorithm not launch" << std::endl;
	}
	else if (_iterGlobal < _iterG) {
		std::cout << "method " << _name << " converged in " << _iterGlobal << " iterations." << std::endl;
		std::cout << "Converged in " << (float)tMarket / CLOCKS_PER_SEC << " seconds" << std::endl;

	}
	else {
		std::cout << "method " << _name << " not converged in " << _iterGlobal << " iterations." << std::endl;
		std::cout << "time taken " << (float) tMarket / CLOCKS_PER_SEC << " seconds" << std::endl;
	}
	resF.display();
	std::cout << "The power error of this state is (constraint) " << resF.get(0, _iterGlobal / _stepG - (_iterGlobal>0)) << " and convergence " << resF.get(1, _iterGlobal / _stepG - (_iterGlobal > 0)) << std::endl;
	std::cout << "===============================================================|" << std::endl;
	std::cout << "      System Summary                                           |" << std::endl;
	std::cout << "===============================================================|" << std::endl;
	std::cout << "Agent            " << _nAgentTrue << std::endl;
	std::cout << "Nombre d'�change " << _nTrade << std::endl;

	std::cout << std::endl << std::endl;

	std::cout << "==================================================================================================|" << std::endl;
	std::cout << "      Agent Data                                                                                  |" << std::endl;
	std::cout << "==================================================================================================|" << std::endl;
	std::cout << " Agent |  Cost   |  Cost   |        Power Injection          |           Power Injection          |" << std::endl;
	std::cout << "  #    |  a (pu) |  b (pu) |  P (pu) | Pmin (pu) | Pmax (pu) |  Q (pu)    | Qmin (pu) | Qmax (pu) |" << std::endl;
	std::cout << "-------|---------|---------|---------|-----------|-----------|------------|-----------|-----------|" << std::endl;

	for (int n = 0; n < _nAgentTrue; n++) {
		
		std::cout << std::setw(7) << n << "|" << std::setw(8) << a.get(n, 0) << " |" << std::setw(9)
			<< b.get(n, 0) << "|" << std::setw(9) << Pn.get(n, 0) << "|" << std::setw(11)
			<< Pmin.get(n, 0) * nVoisin.get(n,0) << "|" << std::setw(11) << Pmax.get(n, 0) * nVoisin.get(n, 0) << "|";
		if (isAC) {
			std::cout << std::setw(12) << Pn.get(n + _nAgentTrue, 0) << "|" << std::setw(11)
				<< Pmin.get(n + _nAgentTrue, 0) * nVoisin.get(n + _nAgentTrue, 0) << "|" << std::setw(11) << Pmax.get(n + _nAgentTrue, 0) * nVoisin.get(n + _nAgentTrue, 0) << "|" << std::endl;
		}
		else {
			std::cout << std::setw(10) << 0 << "|" << std::setw(11)
				<< 0 << "|" << std::setw(11) << 0 << "|" << std::endl;
		}
		
	}


	std::cout << "========================================================================================================|" << std::endl;
	std::cout << "                      END PRINT                                                                         |" << std::endl;
	std::cout << "========================================================================================================|" << std::endl;


	std::cout << std::endl << std::endl;
}




__global__ void setMinPowerForSolve(float* Pmax, float* Pmin, int nCons) {

	int thI = threadIdx.x + blockDim.x * blockIdx.x;
	int step = gridDim.x * blockDim.x;

	for (int i = thI; i < nCons; i += step) {
		Pmax[i] = Pmin[i];
	}

}
