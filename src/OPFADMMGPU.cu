#include "hip/hip_runtime.h"
#include "../head/OPFADMMGPU.cuh"
 
#define NMAXAGENTPERTHREAD 5

OPFADMMGPU::OPFADMMGPU() : MethodOPFGPU()
{
#if DEBUG_CONSTRUCTOR
	std::cout << " OPFADMMGPU Constructor" << std::endl;
#endif // DEBUG_CONSTRUCTOR
	_name = NAME;
	timePerBlock = MatrixCPU(1, 12, 0); // Fb0, Fb11abcd, FB12, Fb2, Fb3, Fb4, Fb5,FB6, Fb0'
	// si les sous ensemble ne sont pas accessible, tout est dans le premier.
	occurencePerBlock = MatrixCPU(1, 12, 0); //nb de fois utilisé pendant la simu
}




OPFADMMGPU::OPFADMMGPU(float rho) : MethodOPFGPU()
{
#if DEBUG_CONSTRUCTOR
	std::cout << "default OPFADMMGPU Constructor" << std::endl;
#endif // DEBUG_CONSTRUCTOR
	_name = NAME;
	_rho = rho;
	timePerBlock = MatrixCPU(1, 12, 0); // Fb0, Fb11, FB12, Fb2, Fb3, Fb4, Fb5,FB6, Fb0'
	// si les sous ensemble ne sont pas accessible, tout est dans le premier.
	occurencePerBlock = MatrixCPU(1, 12, 0); //nb de fois utilisé pendant la simu
}

OPFADMMGPU::~OPFADMMGPU()
{
	/*DELETEA(tempM1);
	DELETEA(tempM);

	DELETEA(X);
	DELETEA(Ypre);
	DELETEA(Y);
	DELETEA(YTrans);
	DELETEA(Mu);

	DELETEA(Hinv);
	DELETEA(A);
	DELETEA(Q);

	DELETEA(Childs);*/

}
void OPFADMMGPU::setParam(float rho)
{
	_rho = rho;
}

bool OPFADMMGPU::chekcase()
{
	if (_nBus != (_nLine + 1)) {
		std::cout << "wrong number of line " << _nLine << "against " << _nBus << std::endl;
		return false;
	}
	for (int i = 0; i < _nLine; i++) {
		if (CoresLineBus.get(i, 1) != (i + 1)) {
			std::cout << "wrong numerotation of line " << CoresLineBus.get(i, 1) << "against " << (i + 1) << std::endl;
			return false;
		}
		if (CoresLineBus.get(i, 0) > CoresLineBus.get(i, 1)) {
			std::cout << "wrong numeoration of bus " << CoresLineBus.get(i, 0) << "against " << CoresLineBus.get(i, 1) << std::endl;
			return false;
		}
	}
	if (ZsRe.getNLin() == 0 || ZsIm.getNLin() == 0) {
		std::cout << "matrice non defined, ZsRe, Zs Im, Yd" << std::endl;
		ZsRe.display();
		ZsIm.display();
		return false;
	}

	return true;
}

void OPFADMMGPU::solve(Simparam* result, const Simparam& sim, const StudyCase& cas)
{
#ifdef DEBUG_SOLVE
	cas.display();
	sim.display(1);
#endif // DEBUG_SOLVE

	clock_t tall = clock();
#ifdef INSTRUMENTATION
	std::chrono::high_resolution_clock::time_point t1;
	std::chrono::high_resolution_clock::time_point t2;
#endif // INSTRUMENTATION

	// FB 0
	if (_id == 0) {
#ifdef INSTRUMENTATION
		t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION
		init(sim, cas);
#ifdef INSTRUMENTATION
		hipDeviceSynchronize();
		t2 = std::chrono::high_resolution_clock::now();
		timePerBlock.increment(0, 0, (float) std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
		occurencePerBlock.increment(0, 0, 1);
#endif // INSTRUMENTATION
	}

	_iterG = sim.getIterG();
	int iterL = sim.getIterL();
	_stepG = sim.getStepG();
	int stepL = sim.getStepL();

	float epsG = sim.getEpsG();
	float epsL = MYMIN(sim.getEpsL(), epsG / 200);
	float rhoInit = sim.getRho();


	float fc = 0;
	float resG = 2 * epsG;
	float resL = 2 * epsL;
	_iterGlobal = 0;

	//Chat.display(true);
	//

	/*CoresSoloBusAgent.display(true);
	Apt1.display(true);
	Apt2.display(true);
	Bpt2.display(true);
	Cost1.display(true);
	Cost2.display(true);
	Pmin.display(true);
	Pmax.display(true);

	PnPre.display(true);
	PnMoy.display(true);
	PnTilde.display(true);
	MuL.display(true);
	_nAgentByBus.display(true);
	std::cout << _rhol << " " << epsL << " " << iterL << " " << _nAgent << " " << _nBus;
	_CoresAgentBus.display(true); 
	_CoresAgentBusBegin.display(true);

	std::cout << "------" << std::endl;*/

	while ((_iterGlobal < _iterG) && (resG > epsG)) {

		/*std::cout << "--------" << std::endl;
		std::cout << " Pn " << std::endl;
		Pn.display(true);
		std::cout << " PnTilde " << std::endl;
		PnTilde.display(true);
		std::cout << " X " << std::endl;
		X.display(true);



		std::cout << " Y " << std::endl;
		Y.display(true);
		std::cout << " Mu " << std::endl;
		Mu.display(true);
		Chat.display(true);

*/
#ifdef INSTRUMENTATION
		t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION

		updateLocalProb(epsL, iterL);
		//CHECK_LAST_CUDA_ERROR();
		/*std::cout << " Pn " << std::endl;
		Pn.display(true);
		std::cout << " PnMoy " << std::endl;
		PnMoy.display(true);
		std::cout << " PnTilde " << std::endl;
		PnTilde.display(true);
		std::cout << " MuL " << std::endl;
		MuL.display(true);*/
#ifdef INSTRUMENTATION
		hipDeviceSynchronize();
		t2 = std::chrono::high_resolution_clock::now();
		timePerBlock.increment(0, 1, (float) std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
		t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION

		//std::cout << _numBlocksB << " " << _blockSize << std::endl;
		updateXOPFADMM << <_numBlocksB, _blockSize >> > (X._matrixGPU, Chat._matrixGPU, VoltageLimit._matrixGPU, PnTilde._matrixGPU, _nAgentByBus._matrixGPU, nChild._matrixGPU, _indiceBusBegin._matrixGPU, _nBus, Lagrange);
		//CHECK_LAST_CUDA_ERROR();
#ifdef INSTRUMENTATION
		hipDeviceSynchronize();
		t2 = std::chrono::high_resolution_clock::now();
		timePerBlock.increment(0, 5, (float) std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
		t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION

		CommunicationX();
		//CHECK_LAST_CUDA_ERROR();
#ifdef INSTRUMENTATION
		hipDeviceSynchronize();
		t2 = std::chrono::high_resolution_clock::now();
		timePerBlock.increment(0, 6, (float) std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
		t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION

		updateGlobalProb();
		updateMu();
		//CHECK_LAST_CUDA_ERROR();
#ifdef INSTRUMENTATION
		hipDeviceSynchronize();
		t2 = std::chrono::high_resolution_clock::now();
		timePerBlock.increment(0, 7, (float) std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
		t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION

		updateChat();
		CHECK_LAST_CUDA_ERROR();
#ifdef INSTRUMENTATION
		hipDeviceSynchronize();
		t2 = std::chrono::high_resolution_clock::now();
		timePerBlock.increment(0, 8, (float) std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
#endif // INSTRUMENTATION
		// FB 4
		if (!(_iterGlobal % _stepG)) {
#ifdef INSTRUMENTATION
			t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION
			resG = updateRes(_iterGlobal / _stepG);
			//std::cout << _iterGlobal << " " << _iterLocal << " " << _rho << " " << resL << " " << resF.get(0, _iterGlobal / _stepG) << " " << resF.get(1, _iterGlobal / _stepG) << std::endl;
			//resG = 1;
#ifdef INSTRUMENTATION
			hipDeviceSynchronize();
			t2 = std::chrono::high_resolution_clock::now();
			timePerBlock.increment(0, 9, (float) std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
#endif // INSTRUMENTATION
		}
		//std::cout << iterGlobal << " " << _iterLocal << " " << resL << " " << resF.get(0, iterGlobal / stepG) << " " << resF.get(1, iterGlobal / stepG) << std::endl;

		_iterGlobal++;
	}
	//std::cout << iterGlobal << " " << _iterLocal << " " << resL << " " << resF.get(0, (iterGlobal - 1) / stepG) << " " << resF.get(1, (iterGlobal - 1) / stepG) << " " << resF.get(2, (iterGlobal - 1) / stepG) << std::endl;


#ifdef INSTRUMENTATION	
	occurencePerBlock.increment(0, 1, _iterGlobal);
	occurencePerBlock.increment(0, 5, _iterGlobal);
	occurencePerBlock.increment(0, 6, _iterGlobal);
	occurencePerBlock.increment(0, 7, _iterGlobal);
	occurencePerBlock.increment(0, 8, _iterGlobal);
	occurencePerBlock.increment(0, 9, _iterGlobal / _stepG);

	t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION

	//std::cout << "--------" << std::endl;
	/*std::cout << " Pn " << std::endl;
	Pn.display(true);
	std::cout << " PnTilde " << std::endl;
	PnTilde.display(true);*/

	fc = calcFc(&Cost1, &Cost2, &Pn, &tempN2);
	// FB 5

	result->setResF(&resF);

	/*MatrixGPU Param(1, 12);
	Param.set(0, 0, _nAgent);
	Param.set(0, 1, _nBus);
	Param.set(0, 2, _nLine);
	Param.set(0, 3, rhoInit);
	Param.set(0, 4, _rhol);
	Param.set(0, 5, _stepG);
	Param.set(0, 6, fc);
	Param.set(0, 7, Pn.get(1, 0));
	Param.set(0, 8, Pn.get(2, 0));
	Param.set(0, 9, Pn.get(4, 0));
	Param.set(0, 10, Pn.get(5, 0));
	Param.set(0, 11, _iterGlobal);

	std::string nameFile = "Residuals_RhoConst_W0agent.csv";
	Param.saveCSV(nameFile);
	resF.saveCSV(nameFile);*/

	result->setIter(_iterGlobal);
	MatrixCPU PnCPU;
	Pn.toMatCPU(PnCPU);
	result->setPn(&PnCPU);
	
	MatrixCPU Pb(getPb());
	MatrixCPU Phi(getPhi());
	MatrixCPU E(getE());
	
	result->setE(&E);
	result->setPhi(&Phi);
	result->setPb(&Pb);
	

	result->setFc(fc);

#ifdef INSTRUMENTATION
	hipDeviceSynchronize();
	t2 = std::chrono::high_resolution_clock::now();
	timePerBlock.increment(0, 10, (float) std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
	occurencePerBlock.increment(0, 10, 1);

	result->setTimeBloc(&timePerBlock, &occurencePerBlock);
#endif // INSTRUMENTATION
	tall = clock() - tall;
	timeOPF = tall;

	result->setTime((float)tall / CLOCKS_PER_SEC);

}

void OPFADMMGPU::updateP0(const StudyCase& cas)
{
	_id = _id + 1;
#ifdef INSTRUMENTATION
	std::chrono::high_resolution_clock::time_point t1 = std::chrono::high_resolution_clock::now();
#endif
	Pmin = cas.getPmin();
	Pmax = cas.getPmax();
	Cost2 = cas.getb();

	// pour essayer que cela marche
	Pn.add(&Pmin, &Pmax);
	Pn.divide(2);
	// remove loss agent
	Pn.set(0, 0, 0, 1);
	Pmin.set(0, 0, 0, 1);
	Pmax.set(0, 0, 0, 1);
	Pn.set(_nAgent, 0, 0, 1);
	Pmin.set(_nAgent, 0, 0, 1);
	Pmax.set(_nAgent, 0, 0, 1);

	ComputePFromAgentToBus();
	divideMultiplyByNagentByBus << <_numBlocksB, _blockSize >> > (Apt1._matrixGPU, Apt2._matrixGPU, PnTilde._matrixGPU, PnTmin._matrixGPU, PnTmax._matrixGPU, _nAgentByBus._matrixGPU, _rhol, _nBus);
	PnMoy.set(&PnTilde);
	initPQ << < _numBlocksB, _blockSize >> > (X._matrixGPU, _indiceBusBegin._matrixGPU, _nAgentByBus._matrixGPU, PnTilde._matrixGPU, _nBus);
	initDFSPQ << <1, _nBus, _nBus* (sizeof(bool) + sizeof(int)) >> > (X._matrixGPU, nChild._matrixGPU, Childs._matrixGPU, _indiceBusBegin._matrixGPU, _indiceChildBegin._matrixGPU, _nBus);
	communicateX << <_nBus, _blockSize >> > (X._matrixGPU, nChild._matrixGPU, Ancestor._matrixGPU, Childs._matrixGPU, _indiceBusBegin._matrixGPU, _indiceChildBegin._matrixGPU, _nBus);


	Y.set(&X);
#ifdef INSTRUMENTATION
	hipDeviceSynchronize();
	std::chrono::high_resolution_clock::time_point t2 = std::chrono::high_resolution_clock::now();
	timePerBlock.increment(0, 11, (float) std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
	occurencePerBlock.increment(0, 11, 1);
	t1 = std::chrono::high_resolution_clock::now();
#endif

	updateChat();
#ifdef INSTRUMENTATION
	hipDeviceSynchronize();
	t2 = std::chrono::high_resolution_clock::now();
	timePerBlock.increment(0, 8, (float) std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
	occurencePerBlock.increment(0, 8, 1);

#endif


}

void OPFADMMGPU::init(const Simparam& sim, const StudyCase& cas)
{
	// intitilisation des matrixs et variables 

	clock_t t = clock();
	//std::cout << "init " << std::endl;
	_rho = sim.getRho();

	if (_rhol == 0) {
		_rhol = _rho;
	}
	if (consensus) {
		std::cout << "pas coder pour update Q !!!" << std::endl;
		exit(-1);
	}

	const int iterG = sim.getIterG();
	const int stepG = sim.getStepG();

	_nAgent = cas.getNagent();

	_nBus = cas.getNBus();
	_nLine = cas.getNLine(true); // ne doit pas �tre r�duit ici !!!
	_sizeOPFTotal = 3 * _nLine + 7 * _nBus; // L = nChild.sum()
	_numBlocksN = ceil((_nAgent + _blockSize - 1) / _blockSize);
	_numBlocksM = ceil((_sizeOPFTotal + _blockSize - 1) / _blockSize);
	_numBlocksB = ceil((_nBus + _blockSize - 1) / _blockSize);
	_nAgentByBus = MatrixGPU(cas.getNagentByBus(), 1);
	_nAgentByBus.preallocateReduction();
	// 2 sources d'erreurs il faut que 
	int nAgentMaByBus = _nAgentByBus.max2();
	if (nAgentMaByBus > _blockSizeSmall / 2) {
		throw std::invalid_argument("the number of agent by bus is too high , must change blocksize if possible");
	}

	// il faut remettre sur CPU ce qu'il faut !!!
	if (tempL.getPos()) {
		Ancestor.transferCPU();
		_indiceBusBegin.transferCPU();
		PosChild.transferCPU();
		_indiceChildBegin.transferCPU();
		Childs.transferCPU();
		tempL.transferCPU();
	}

	tempL = MatrixGPU(_nLine, 1);


	//std::cout << _nAgent << " " << _nBus << " " << _nLine << std::endl;

	nChildCPU = MatrixCPU(_nBus, 1);
	CoresLineBus = cas.getCoresLineBus();
	_CoresBusAgent = cas.getCoresBusAgentLin(); // Cores[n] = b
	Ancestor = MatrixGPU(_nBus, 1, 0); // A_i = bus ant�c�dent de i
	Ancestor.set(0, 0, -1); // the slack bus has no ancestor
	ZsRe = MatrixGPU(cas.getZsRe());
	ZsIm = MatrixGPU(cas.getZsImag());
	ZsNorm = MatrixGPU(_nLine, 1);
	ZsNorm.multiplyT(&ZsRe, &ZsRe);
	tempL.multiplyT(&ZsIm, &ZsIm);
	ZsNorm.add(&tempL);
	tempL.transferGPU();



	if (!chekcase()) {
		throw std::invalid_argument("not a radial case");
	}

	for (int lold = 0; lold < _nLine; lold++) {
		int l = lold + 1;
		int busTo = l;
		int busFrom = CoresLineBus.get(lold, 0);
		Ancestor.set(busTo, 0, busFrom);
		nChildCPU.set(busFrom, 0, nChildCPU.get(busFrom, 0) + 1); // pas parallelisable -> reduction chelou
	}

	nChild = MatrixGPU(nChildCPU, 1);




	_rhoInv = 1 / _rho;
	resF = MatrixCPU(3, (iterG / stepG) + 1, 0);


	MatrixGPU lowerBound(cas.getLowerBound(), 1); //voltage angle, voltage, line...
	MatrixGPU upperBound(cas.getUpperBound(), 1); //voltage angle, voltage, line...


	//std::cout << " local resolution " << std::endl;
	// local resolution
	tempN2 = MatrixGPU(2 * _nAgent, 1, 0, 1);
	tempB2 = MatrixGPU(2 * _nBus, 1, 0, 1);
	CoresSoloBusAgent = MatrixGPU(_nBus, 1, -1, 1);
	Pn = MatrixGPU(sim.getPn(), 1);
	Pn.preallocateReduction();
	Pmin = MatrixGPU(cas.getPmin(), 1);
	Pmax = MatrixGPU(cas.getPmax(), 1);


	PnTmin = MatrixGPU(2 * _nBus, 1, 0, 1);
	PnTmax = MatrixGPU(2 * _nBus, 1, 0, 1);

	Cost1 = MatrixGPU(cas.geta(), 1);
	Cost2 = MatrixGPU(cas.getb(), 1);


	PnMoy = MatrixGPU(2 * _nBus, 1, 0, 1);
	PnPre = MatrixGPU(sim.getPn(), 1);
	MuL = MatrixGPU(2 * _nBus, 1, 0, 1);
	PnTilde = MatrixGPU(2 * _nBus, 1, 0, 1);
	Bp1 = MatrixGPU(2 * _nAgent, 1, 0, 1);
	Bpt1 = MatrixGPU(2 * _nBus, 1, 0, 1);
	Bpt2 = MatrixGPU(2 * _nBus, 1, 0, 1);
	Apt1 = MatrixGPU(2 * _nBus, 1, 0, 1);
	Apt2 = MatrixGPU(2 * _nBus, 1, 0, 1);


	if (Pn.max2() < 0.00001) {
		Pn.add(&Pmin, &Pmax);
		Pn.divide(2);
	}
	_CoresAgentBus = MatrixGPU(cas.getCoresAgentBusLin(), 1);
	_CoresAgentBusBegin = MatrixGPU(cas.getCoresAgentBusLinBegin(), 1);

	// remove the grid agent


	Pn.set(0, 0, 0, 1);
	Pmin.set(0, 0, 0, 1);
	Pmax.set(0, 0, 0, 1);
	Pn.set(_nAgent, 0, 0, 1);
	Pmin.set(_nAgent, 0, 0, 1);
	Pmax.set(_nAgent, 0, 0, 1);



	//std::cout << "remove loss agent" << std::endl;
	removeLossAgent << <1, 1 >> > (_nAgentByBus._matrixGPU, _CoresAgentBusBegin._matrixGPU);


	ComputePFromAgentToBus();


	divideMultiplyByNagentByBus << <_numBlocksB, _blockSize >> > (Apt1._matrixGPU, Apt2._matrixGPU, PnTilde._matrixGPU, PnTmin._matrixGPU, PnTmax._matrixGPU, _nAgentByBus._matrixGPU, _rhol, _nBus);


	PnMoy.set(&PnTilde);
	
	
	/*std::cout << "Apt1 " << std::endl;
	Apt1.display(true);
	std::cout << "Apt2 " << std::endl;
	Apt2.display(true);
	std::cout << " Pmin " << std::endl;
	Pmin.display(true);
	std::cout << " Pmax " << std::endl;
	Pmax.display(true);

	std::cout << " Pn  limits " << std::endl;
	PnTmin.display(true);
	PnTmax.display(true);*/
	


	//std::cout << " creation " << std::endl;
	X = MatrixGPU(_sizeOPFTotal, 1, 0, 1); // Changement d'ordre !!!!!!!!!!!!
	Ypre = MatrixGPU(_sizeOPFTotal, 1, 0, 1); // (Pi, Qi, li, vi, pi, qi, vai, Pci ..., Qci... , lci...) !!!!!
	Y = MatrixGPU(_sizeOPFTotal, 1, 0, 1);
	Y.preallocateReduction();
	//YTrans = MatrixGPU(_sizeOPFTotal, 1, 0, 1);
	Mu = MatrixGPU(_sizeOPFTotal, 1, 0, 1);

	tempN1 = MatrixGPU(_nAgent, 1);
	tempNN = MatrixGPU(_nAgent, _nAgent);
	//tempM1 = new MatrixGPU[_nAgent];
	tempM = MatrixGPU(_sizeOPFTotal, 1, 0, 1);
	sizeOPFADMMGPU = MatrixGPU(_nBus, 1, 0, 1);
	sizeOPFADMMGPU.preallocateReduction();
	sizeOPFADMMGPUBig = MatrixGPU(_sizeOPFTotal, 1, 0, 1);
	_indiceBusBegin = MatrixGPU(_nBus, 1);
	_indiceBusBeginBig = MatrixGPU(_sizeOPFTotal, 1, 0, 1);
	int debut = 0;
	for (int i = 0; i < _nBus; i++) {
		int m = nChildCPU.get(i, 0);
		_indiceBusBegin.set(i, 0, debut);
		int sizeA = m * 3 + 7;
		debut += sizeA;
	}
	_indiceBusBegin.transferGPU();
	defineSizeBig << <_nBus, _blockSize >> > (sizeOPFADMMGPUBig._matrixGPU, nChild._matrixGPU, _indiceBusBegin._matrixGPU, sizeOPFADMMGPU._matrixGPU, _indiceBusBeginBig._matrixGPU);
	/*sizeOPFADMMGPU = nChild;
	sizeOPFADMMGPU.multiply(3);
	sizeOPFADMMGPU.add(7);
	sizeOPFADMMGPU.display(true);
	sizeOPFADMMGPUBig.display(true);*/

	_sizeOPFMax = sizeOPFADMMGPU.max2();
	Hinv = MatrixGPU(_sizeOPFTotal, _sizeOPFMax, 0, 1);
	Q = MatrixGPU(_sizeOPFTotal, 1, 0, 1);

	Childs = MatrixGPU(_nLine, 1);
	PosChild = MatrixGPU(_nBus, 1, -1);
	Chat = MatrixGPU(6, _nBus, 0, 1);
	VoltageLimit = MatrixGPU(2, _nBus, 0, 1); // min, max
	VoltageLimitReal = MatrixGPU(2, _nBus, 0, 1); // min, max


	_indiceChildBegin = MatrixGPU(_nLine, 1);
	//int sizeOPF2 = 1 * nChild.get(i, 0) + 9;


	MatrixCPU nChildTemp(_nBus, 1, 0);
	//lowerBound.display(true);
	//upperBound.display(true);
	initVoltageBound << < _numBlocksB, _blockSize >> > (VoltageLimitReal._matrixGPU, VoltageLimit._matrixGPU, lowerBound._matrixGPU, upperBound._matrixGPU, nChild._matrixGPU, _nBus);

	//nChild.display(true);
	//VoltageLimit.display(true);
	//VoltageLimitReal.display(true);
	//

	//nChild.display();
	//std::cout << " Child " << std::endl;
	int debutChild = 0;
	for (int i = 0; i < _nBus; i++) {
		if (i > 0) {
			_indiceChildBegin.set(i - 1, 0, debutChild);

			int Ai = Ancestor.get(i, 0);
			Childs.set(_indiceChildBegin.get(Ai, 0) + nChildTemp.get(Ai, 0), 0, i);
			PosChild.set(i, 0, nChildTemp.get(Ai, 0));
			nChildTemp.increment(Ai, 0, 1);
			debutChild += nChildCPU.get(i - 1, 0);
		}

	}
	/*Childs.display();
	Ancestor.display();
	nChildCPU.display();
	PosChild.display();
	std::cout << " _indiceChildBegin " << std::endl;
	_indiceChildBegin.display(true);*/
	Ancestor.transferGPU();
	PosChild.transferGPU();
	debut = 0;
	//std::cout << " Hinv " << std::endl;
	for (int i = 0; i < _nBus; i++) {
		int m = nChildCPU.get(i, 0);
		int sizeA = nChildCPU.get(i, 0) * 3 + 7;
		MatrixCPU A(2 + (i > 0), sizeA);

		if (i > 0) {
			A.set(2, 0, 2 * ZsRe.get(i - 1, 0));
			A.set(2, 1, 2 * ZsIm.get(i - 1, 0));
			A.set(2, 2, -ZsNorm.get(i - 1, 0));
			A.set(2, 3, -1);
			A.set(2, 6, 1);
			A.set(0, 0, -1);
			A.set(1, 1, -1);
		}
		A.set(0, 4, 1);
		A.set(1, 5, 1);

		for (int j = 0; j < m; j++) {
			int c = Childs.get(_indiceChildBegin.get(i, 0) + j, 0);
			A.set(0, 7 + j, 1); // Pci
			A.set(1, 7 + m + j, 1); // Qci
			A.set(0, 7 + 2 * m + j, -ZsRe.get(c - 1, 0)); // -R l
			A.set(1, 7 + 2 * m + j, -ZsIm.get(c - 1, 0)); // -X l
		}

		//A[i].display();

		MatrixCPU temp33(2 + 1 * (i > 0), 2 + 1 * (i > 0));
		MatrixCPU temp3M(2 + 1 * (i > 0), sizeA);
		MatrixCPU tempMM(sizeA, sizeA);
		MatrixCPU tempMMbis(sizeA, sizeA);

		temp33.multiplyTrans(&A, &A);
		temp33.invertGaussJordan(&temp33);
		temp3M.MultiplyMatMat(&temp33, &A);
		tempMM.multiplyTrans(&A, &temp3M, 0);

		tempMMbis.setEyes(-1);
		tempMMbis.add(&tempMM);
		MatrixGPU tempMMGPU = MatrixGPU(tempMMbis, 1);
		Hinv.setBloc(debut, debut + sizeA, 0, sizeA, &tempMMGPU);
		debut += sizeA;
	}
	Hinv.divide(_rho);
	//Hinv.display(true);
	_indiceChildBegin.transferGPU();
	Childs.transferGPU();
	//std::cout << " Childs " << std::endl;
	//Childs.display(true);
	//std::cout << " init valeur " << std::endl;

	initPQV << < _numBlocksB, _blockSize >> > (X._matrixGPU, _indiceBusBegin._matrixGPU, _nAgentByBus._matrixGPU, PnTilde._matrixGPU, _nBus);
	/*std::cout << " X " << std::endl;
	X.display(true);
	std::cout << " _indiceBusBegin " << std::endl;
	_indiceBusBegin.display(true);
	std::cout << " _indiceChildBegin " << std::endl;
	_indiceChildBegin.display(true);
	std::cout << " Childs " << std::endl;
	Childs.display(true);
	std::cout << " nChild " << std::endl;
	nChild.display(true);
	std::cout << " posChild " << std::endl;
	PosChild.display(true);*/
	//
	initDFSPQ << <1, _nBus, _nBus* (sizeof(bool) + sizeof(int)) >> > (X._matrixGPU, nChild._matrixGPU, Childs._matrixGPU, _indiceBusBegin._matrixGPU, _indiceChildBegin._matrixGPU, _nBus);




	//CHECK_LAST_CUDA_ERROR();

	communicateX << <_nBus, _blockSize >> > (X._matrixGPU, nChild._matrixGPU, Ancestor._matrixGPU, Childs._matrixGPU, _indiceBusBegin._matrixGPU, _indiceChildBegin._matrixGPU, _nBus);


	Y.set(&X);
	/*std::cout << " X " << std::endl;
	X.display(true);
	std::cout << " Y " << std::endl;
	Y.display(true);
	std::cout << " Q " << std::endl;
	Q.display(true);
	std::cout << " Mu "<< std::endl;
	Mu.display(true);*/

	updateChat();
	/*std::cout << " Chat " << std::endl;
	Chat.display(true);
	std::cout << " Bpt2 " << std::endl;
	Bpt2.display(true);
	std::cout << " Cp " << std::endl;
	Cost2.display(true);
	std::cout << " Ap2 " << std::endl;
	Cost1.display(true);
	std::cout << " Nagent " << std::endl;
	_nAgentByBus.display(true);
	std::cout << " Bus Agent : agent->bus " << std::endl;
	_CoresBusAgent.display(true);
	std::cout << " Agent bus : bus->agent " << std::endl;
	_CoresAgentBus.display(true);
	std::cout << " Agent bus begin : bus->agent " << std::endl;
	_CoresAgentBusBegin.display(true);

	std::cout << "Apt2 " << std::endl;
	Apt2.display(true);
	std::cout << " Pmin " << std::endl;
	Pmin.display(true);
	std::cout << " Pma " << std::endl;
	Pmax.display(true);
	std::cout << " CoresSoloBusAgent " << std::endl;
	CoresSoloBusAgent.display(true);*/
	//std::cout << "fin init temps : " << (float)(clock() - t) / CLOCKS_PER_SEC << std::endl;
	//std::cout << "---------------------------------------------------------------------------------------" << std::endl;
	CHECK_LAST_CUDA_ERROR();
}

void OPFADMMGPU::updateGlobalProb() {

	Ypre.swap(&Y);
	int numBlock = _sizeOPFTotal;
	switch (_blockSize) {
	case 512:
		updateY<512> << <numBlock, _blockSize >> > (Y._matrixGPU, Hinv._matrixGPU, Q._matrixGPU, sizeOPFADMMGPUBig._matrixGPU, _indiceBusBeginBig._matrixGPU, _sizeOPFMax);
		break;
	case 256:
		updateY<256> << <numBlock, _blockSize >> > (Y._matrixGPU, Hinv._matrixGPU, Q._matrixGPU, sizeOPFADMMGPUBig._matrixGPU, _indiceBusBeginBig._matrixGPU, _sizeOPFMax);
		break;
	case 128:
		updateY<128> << <numBlock, _blockSize >> > (Y._matrixGPU, Hinv._matrixGPU, Q._matrixGPU, sizeOPFADMMGPUBig._matrixGPU, _indiceBusBeginBig._matrixGPU, _sizeOPFMax);
		break;
	case 64:
		updateY< 64> << <numBlock, _blockSize >> > (Y._matrixGPU, Hinv._matrixGPU, Q._matrixGPU, sizeOPFADMMGPUBig._matrixGPU, _indiceBusBeginBig._matrixGPU, _sizeOPFMax);
		break;
	case 32:
		updateY< 32> << <numBlock, _blockSize >> > (Y._matrixGPU, Hinv._matrixGPU, Q._matrixGPU, sizeOPFADMMGPUBig._matrixGPU, _indiceBusBeginBig._matrixGPU, _sizeOPFMax);
		break;
	case 16:
		updateY< 16> << <numBlock, _blockSize >> > (Y._matrixGPU, Hinv._matrixGPU, Q._matrixGPU, sizeOPFADMMGPUBig._matrixGPU, _indiceBusBeginBig._matrixGPU, _sizeOPFMax);
		break;
	case  8:
		updateY<  8> << <numBlock, _blockSize >> > (Y._matrixGPU, Hinv._matrixGPU, Q._matrixGPU, sizeOPFADMMGPUBig._matrixGPU, _indiceBusBeginBig._matrixGPU, _sizeOPFMax);
		break;
	case  4:
		updateY<  4> << <numBlock, _blockSize >> > (Y._matrixGPU, Hinv._matrixGPU, Q._matrixGPU, sizeOPFADMMGPUBig._matrixGPU, _indiceBusBeginBig._matrixGPU, _sizeOPFMax);
		break;
	case  2:
		updateY<  2> << <numBlock, _blockSize >> > (Y._matrixGPU, Hinv._matrixGPU, Q._matrixGPU, sizeOPFADMMGPUBig._matrixGPU, _indiceBusBeginBig._matrixGPU, _sizeOPFMax);
		break;
	case  1:
		updateY<  1> << <numBlock, _blockSize >> > (Y._matrixGPU, Hinv._matrixGPU, Q._matrixGPU, sizeOPFADMMGPUBig._matrixGPU, _indiceBusBeginBig._matrixGPU, _sizeOPFMax);
		break;
	}

	Y.set(3, 0, 1, 1);
	Y.set(6, 0, 1, 1);

}


void OPFADMMGPU::solveConsensus(float eps, MatrixCPU* PSO)
{
	throw std::invalid_argument("WIP !!");
}

void OPFADMMGPU::initConsensus(const Simparam& sim, const StudyCase& cas, float rhoSO)
{
	throw std::invalid_argument("WIP !!");
}

void OPFADMMGPU::updateConsensus(MatrixCPU* Pmarket)
{
	throw std::invalid_argument("WIP !!");
}

void OPFADMMGPU::solveConsensus(float eps, MatrixGPU* PSO)
{
	throw std::invalid_argument("WIP !!");
}

void OPFADMMGPU::updateConsensus(MatrixGPU* Pmarket)
{
	throw std::invalid_argument("WIP !!");
}


void OPFADMMGPU::updateLocalProb(float epsL, int nIterL) {

	int numBlocks = _nBus;
	switch (_blockSizeSmall) {
	case 512:
		updatePnPGPUSharedResidual<512> << <numBlocks, _blockSizeSmall >> > (Pn._matrixGPU, PnPre._matrixGPU, PnMoy._matrixGPU, PnTilde._matrixGPU, MuL._matrixGPU, _nAgentByBus._matrixGPU, _rhol, Cost1._matrixGPU, Cost2._matrixGPU,
			Pmin._matrixGPU, Pmax._matrixGPU, Apt1._matrixGPU, Apt2._matrixGPU, Bpt2._matrixGPU, CoresSoloBusAgent._matrixGPU, _CoresAgentBus._matrixGPU, _CoresAgentBusBegin._matrixGPU, epsL, nIterL, _nAgent, _nBus);
		break;
	case 256:
		updatePnPGPUSharedResidual<256> << <numBlocks, _blockSizeSmall >> > (Pn._matrixGPU, PnPre._matrixGPU, PnMoy._matrixGPU, PnTilde._matrixGPU, MuL._matrixGPU, _nAgentByBus._matrixGPU, _rhol, Cost1._matrixGPU, Cost2._matrixGPU,
			Pmin._matrixGPU, Pmax._matrixGPU, Apt1._matrixGPU, Apt2._matrixGPU, Bpt2._matrixGPU, CoresSoloBusAgent._matrixGPU, _CoresAgentBus._matrixGPU, _CoresAgentBusBegin._matrixGPU, epsL, nIterL, _nAgent, _nBus);
		break;
	case 128:
		updatePnPGPUSharedResidual<128> << <numBlocks, _blockSizeSmall >> > (Pn._matrixGPU, PnPre._matrixGPU, PnMoy._matrixGPU, PnTilde._matrixGPU, MuL._matrixGPU, _nAgentByBus._matrixGPU, _rhol, Cost1._matrixGPU, Cost2._matrixGPU,
			Pmin._matrixGPU, Pmax._matrixGPU, Apt1._matrixGPU, Apt2._matrixGPU, Bpt2._matrixGPU, CoresSoloBusAgent._matrixGPU, _CoresAgentBus._matrixGPU, _CoresAgentBusBegin._matrixGPU, epsL, nIterL, _nAgent, _nBus);
		break;
	case 64:
		updatePnPGPUSharedResidual< 64> << <numBlocks, _blockSizeSmall >> > (Pn._matrixGPU, PnPre._matrixGPU, PnMoy._matrixGPU, PnTilde._matrixGPU, MuL._matrixGPU, _nAgentByBus._matrixGPU, _rhol, Cost1._matrixGPU, Cost2._matrixGPU,
			Pmin._matrixGPU, Pmax._matrixGPU, Apt1._matrixGPU, Apt2._matrixGPU, Bpt2._matrixGPU, CoresSoloBusAgent._matrixGPU, _CoresAgentBus._matrixGPU, _CoresAgentBusBegin._matrixGPU, epsL, nIterL, _nAgent, _nBus);
		break;
	case 32:
		updatePnPGPUSharedResidual< 32> << <numBlocks, _blockSizeSmall >> > (Pn._matrixGPU, PnPre._matrixGPU, PnMoy._matrixGPU, PnTilde._matrixGPU, MuL._matrixGPU, _nAgentByBus._matrixGPU, _rhol, Cost1._matrixGPU, Cost2._matrixGPU,
			Pmin._matrixGPU, Pmax._matrixGPU, Apt1._matrixGPU, Apt2._matrixGPU, Bpt2._matrixGPU, CoresSoloBusAgent._matrixGPU, _CoresAgentBus._matrixGPU, _CoresAgentBusBegin._matrixGPU, epsL, nIterL, _nAgent, _nBus);
		break;
	case 16:
		updatePnPGPUSharedResidual< 16> << <numBlocks, _blockSizeSmall >> > (Pn._matrixGPU, PnPre._matrixGPU, PnMoy._matrixGPU, PnTilde._matrixGPU, MuL._matrixGPU, _nAgentByBus._matrixGPU, _rhol, Cost1._matrixGPU, Cost2._matrixGPU,
			Pmin._matrixGPU, Pmax._matrixGPU, Apt1._matrixGPU, Apt2._matrixGPU, Bpt2._matrixGPU, CoresSoloBusAgent._matrixGPU, _CoresAgentBus._matrixGPU, _CoresAgentBusBegin._matrixGPU, epsL, nIterL, _nAgent, _nBus);
		break;
	case  8:
		updatePnPGPUSharedResidual<  8> << <numBlocks, _blockSizeSmall >> > (Pn._matrixGPU, PnPre._matrixGPU, PnMoy._matrixGPU, PnTilde._matrixGPU, MuL._matrixGPU, _nAgentByBus._matrixGPU, _rhol, Cost1._matrixGPU, Cost2._matrixGPU,
			Pmin._matrixGPU, Pmax._matrixGPU, Apt1._matrixGPU, Apt2._matrixGPU, Bpt2._matrixGPU, CoresSoloBusAgent._matrixGPU, _CoresAgentBus._matrixGPU, _CoresAgentBusBegin._matrixGPU, epsL, nIterL, _nAgent, _nBus);
		break;
	case  4:
		updatePnPGPUSharedResidual<  4> << <numBlocks, _blockSizeSmall >> > (Pn._matrixGPU, PnPre._matrixGPU, PnMoy._matrixGPU, PnTilde._matrixGPU, MuL._matrixGPU, _nAgentByBus._matrixGPU, _rhol, Cost1._matrixGPU, Cost2._matrixGPU,
			Pmin._matrixGPU, Pmax._matrixGPU, Apt1._matrixGPU, Apt2._matrixGPU, Bpt2._matrixGPU, CoresSoloBusAgent._matrixGPU, _CoresAgentBus._matrixGPU, _CoresAgentBusBegin._matrixGPU, epsL, nIterL, _nAgent, _nBus);
		break;
	case  2:
		updatePnPGPUSharedResidual<  2> << <numBlocks, _blockSizeSmall >> > (Pn._matrixGPU, PnPre._matrixGPU, PnMoy._matrixGPU, PnTilde._matrixGPU, MuL._matrixGPU, _nAgentByBus._matrixGPU, _rhol, Cost1._matrixGPU, Cost2._matrixGPU,
			Pmin._matrixGPU, Pmax._matrixGPU, Apt1._matrixGPU, Apt2._matrixGPU, Bpt2._matrixGPU, CoresSoloBusAgent._matrixGPU, _CoresAgentBus._matrixGPU, _CoresAgentBusBegin._matrixGPU, epsL, nIterL, _nAgent, _nBus);
		break;
	case  1:
		updatePnPGPUSharedResidual<  1> << <numBlocks, _blockSizeSmall >> > (Pn._matrixGPU, PnPre._matrixGPU, PnMoy._matrixGPU, PnTilde._matrixGPU, MuL._matrixGPU, _nAgentByBus._matrixGPU, _rhol, Cost1._matrixGPU, Cost2._matrixGPU,
			Pmin._matrixGPU, Pmax._matrixGPU, Apt1._matrixGPU, Apt2._matrixGPU, Bpt2._matrixGPU, CoresSoloBusAgent._matrixGPU, _CoresAgentBus._matrixGPU, _CoresAgentBusBegin._matrixGPU, epsL, nIterL, _nAgent, _nBus);
		break;
	}
}





void OPFADMMGPU::updateMu()
{
	updateMUGPU << <_numBlocksM, _blockSize >> > (Mu._matrixGPU, Y._matrixGPU, X._matrixGPU, _rho, _sizeOPFTotal);
	/*tempM.subtract(&X, &Y);
	tempM.multiply(_rho);
	if (consensus) {
		tempM.divide(2);
	}
	Mu.add(&tempM);*/

}


float OPFADMMGPU::getPLoss()
{
	float Ploss = 0;
	for (int i = 1; i < _nAgent; i++) {
		Ploss += Pn.get(i, 0);
	}
	return Ploss;
}

float OPFADMMGPU::getQLoss()
{
	float Qloss = 0;
	for (int i = 1; i < _nAgent; i++) {
		Qloss += Pn.get(i + _nAgent, 0);
	}
	return Qloss;
}

void OPFADMMGPU::ComputePFromAgentToBus()
{
	int numBlock = _nBus;
	switch (_blockSize) {
	case 512:
		ComputePFromAgentToBusGPU<512> << <numBlock, _blockSize >> > (PnTilde._matrixGPU, PnTmin._matrixGPU, PnTmax._matrixGPU, CoresSoloBusAgent._matrixGPU, Pn._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, _CoresAgentBus._matrixGPU, _nAgentByBus._matrixGPU, _CoresAgentBusBegin._matrixGPU, _nAgent, _nBus);
		break;
	case 256:
		ComputePFromAgentToBusGPU<256> << <numBlock, _blockSize >> > (PnTilde._matrixGPU, PnTmin._matrixGPU, PnTmax._matrixGPU, CoresSoloBusAgent._matrixGPU, Pn._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, _CoresAgentBus._matrixGPU, _nAgentByBus._matrixGPU, _CoresAgentBusBegin._matrixGPU, _nAgent, _nBus);
		break;
	case 128:
		ComputePFromAgentToBusGPU<128> << <numBlock, _blockSize >> > (PnTilde._matrixGPU, PnTmin._matrixGPU, PnTmax._matrixGPU, CoresSoloBusAgent._matrixGPU, Pn._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, _CoresAgentBus._matrixGPU, _nAgentByBus._matrixGPU, _CoresAgentBusBegin._matrixGPU, _nAgent, _nBus);
		break;
	case 64:
		ComputePFromAgentToBusGPU< 64> << <numBlock, _blockSize >> > (PnTilde._matrixGPU, PnTmin._matrixGPU, PnTmax._matrixGPU, CoresSoloBusAgent._matrixGPU, Pn._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, _CoresAgentBus._matrixGPU, _nAgentByBus._matrixGPU, _CoresAgentBusBegin._matrixGPU, _nAgent, _nBus);
		break;
	case 32:
		ComputePFromAgentToBusGPU< 32> << <numBlock, _blockSize >> > (PnTilde._matrixGPU, PnTmin._matrixGPU, PnTmax._matrixGPU, CoresSoloBusAgent._matrixGPU, Pn._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, _CoresAgentBus._matrixGPU, _nAgentByBus._matrixGPU, _CoresAgentBusBegin._matrixGPU, _nAgent, _nBus);
		break;
	case 16:
		ComputePFromAgentToBusGPU< 16> << <numBlock, _blockSize >> > (PnTilde._matrixGPU, PnTmin._matrixGPU, PnTmax._matrixGPU, CoresSoloBusAgent._matrixGPU, Pn._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, _CoresAgentBus._matrixGPU, _nAgentByBus._matrixGPU, _CoresAgentBusBegin._matrixGPU, _nAgent, _nBus);
		break;
	case  8:
		ComputePFromAgentToBusGPU<  8> << <numBlock, _blockSize >> > (PnTilde._matrixGPU, PnTmin._matrixGPU, PnTmax._matrixGPU, CoresSoloBusAgent._matrixGPU, Pn._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, _CoresAgentBus._matrixGPU, _nAgentByBus._matrixGPU, _CoresAgentBusBegin._matrixGPU, _nAgent, _nBus);
		break;
	case  4:
		ComputePFromAgentToBusGPU<  4> << <numBlock, _blockSize >> > (PnTilde._matrixGPU, PnTmin._matrixGPU, PnTmax._matrixGPU, CoresSoloBusAgent._matrixGPU, Pn._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, _CoresAgentBus._matrixGPU, _nAgentByBus._matrixGPU, _CoresAgentBusBegin._matrixGPU, _nAgent, _nBus);
		break;
	case  2:
		ComputePFromAgentToBusGPU<  2> << <numBlock, _blockSize >> > (PnTilde._matrixGPU, PnTmin._matrixGPU, PnTmax._matrixGPU, CoresSoloBusAgent._matrixGPU, Pn._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, _CoresAgentBus._matrixGPU, _nAgentByBus._matrixGPU, _CoresAgentBusBegin._matrixGPU, _nAgent, _nBus);
		break;
	case  1:
		ComputePFromAgentToBusGPU<  1> << <numBlock, _blockSize >> > (PnTilde._matrixGPU, PnTmin._matrixGPU, PnTmax._matrixGPU, CoresSoloBusAgent._matrixGPU, Pn._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, _CoresAgentBus._matrixGPU, _nAgentByBus._matrixGPU, _CoresAgentBusBegin._matrixGPU, _nAgent, _nBus);
		break;
	}
}

void OPFADMMGPU::updateChat()
{
	int numBlock = _nBus;
	switch (_blockSizeSmall) {
	case 512:
		updateChatGPU<512> << <numBlock, _blockSizeSmall >> > (Chat._matrixGPU, Y._matrixGPU, Mu._matrixGPU, nChild._matrixGPU, Ancestor._matrixGPU, PosChild._matrixGPU, Childs._matrixGPU, _indiceBusBegin._matrixGPU, _indiceChildBegin._matrixGPU, _rho, _nBus);
		break;
	case 256:
		updateChatGPU<256> << <numBlock, _blockSizeSmall >> > (Chat._matrixGPU, Y._matrixGPU, Mu._matrixGPU, nChild._matrixGPU, Ancestor._matrixGPU, PosChild._matrixGPU, Childs._matrixGPU, _indiceBusBegin._matrixGPU, _indiceChildBegin._matrixGPU, _rho, _nBus);
		break;
	case 128:
		updateChatGPU<128> << <numBlock, _blockSizeSmall >> > (Chat._matrixGPU, Y._matrixGPU, Mu._matrixGPU, nChild._matrixGPU, Ancestor._matrixGPU, PosChild._matrixGPU, Childs._matrixGPU, _indiceBusBegin._matrixGPU, _indiceChildBegin._matrixGPU, _rho, _nBus);
		break;
	case 64:
		updateChatGPU< 64> << <numBlock, _blockSizeSmall >> > (Chat._matrixGPU, Y._matrixGPU, Mu._matrixGPU, nChild._matrixGPU, Ancestor._matrixGPU, PosChild._matrixGPU, Childs._matrixGPU, _indiceBusBegin._matrixGPU, _indiceChildBegin._matrixGPU, _rho, _nBus);
		break;
	case 32:
		updateChatGPU< 32> << <numBlock, _blockSizeSmall >> > (Chat._matrixGPU, Y._matrixGPU, Mu._matrixGPU, nChild._matrixGPU, Ancestor._matrixGPU, PosChild._matrixGPU, Childs._matrixGPU, _indiceBusBegin._matrixGPU, _indiceChildBegin._matrixGPU, _rho, _nBus);
		break;
	case 16:
		updateChatGPU< 16> << <numBlock, _blockSizeSmall >> > (Chat._matrixGPU, Y._matrixGPU, Mu._matrixGPU, nChild._matrixGPU, Ancestor._matrixGPU, PosChild._matrixGPU, Childs._matrixGPU, _indiceBusBegin._matrixGPU, _indiceChildBegin._matrixGPU, _rho, _nBus);
		break;
	case  8:
		updateChatGPU<  8> << <numBlock, _blockSizeSmall >> > (Chat._matrixGPU, Y._matrixGPU, Mu._matrixGPU, nChild._matrixGPU, Ancestor._matrixGPU, PosChild._matrixGPU, Childs._matrixGPU, _indiceBusBegin._matrixGPU, _indiceChildBegin._matrixGPU, _rho, _nBus);
		break;
	case  4:
		updateChatGPU<  4> << <numBlock, _blockSizeSmall >> > (Chat._matrixGPU, Y._matrixGPU, Mu._matrixGPU, nChild._matrixGPU, Ancestor._matrixGPU, PosChild._matrixGPU, Childs._matrixGPU, _indiceBusBegin._matrixGPU, _indiceChildBegin._matrixGPU, _rho, _nBus);
		break;
	case  2:
		updateChatGPU<  2> << <numBlock, _blockSizeSmall >> > (Chat._matrixGPU, Y._matrixGPU, Mu._matrixGPU, nChild._matrixGPU, Ancestor._matrixGPU, PosChild._matrixGPU, Childs._matrixGPU, _indiceBusBegin._matrixGPU, _indiceChildBegin._matrixGPU, _rho, _nBus);
		break;
	case  1:
		updateChatGPU<  1> << <numBlock, _blockSizeSmall >> > (Chat._matrixGPU, Y._matrixGPU, Mu._matrixGPU, nChild._matrixGPU, Ancestor._matrixGPU, PosChild._matrixGPU, Childs._matrixGPU, _indiceBusBegin._matrixGPU, _indiceChildBegin._matrixGPU, _rho, _nBus);
		break;
	}

	updateBpt2 << < _numBlocksB, _blockSize >> > (Bpt2._matrixGPU, Chat._matrixGPU, _nAgentByBus._matrixGPU, _nBus);


}

void OPFADMMGPU::CommunicationX()
{
	// X = { Pi, Qi, vi, li, pi, qi, vAi, (Pci, Qci, lci) for all child Ci }

	communicateX << <_nBus, _blockSize >> > (X._matrixGPU, nChild._matrixGPU, Ancestor._matrixGPU, Childs._matrixGPU, _indiceBusBegin._matrixGPU, _indiceChildBegin._matrixGPU, _nBus);

	//Y      (Pi, Qi, vi, li, pi, qi, vai, Pji, Qji, lji)
	updateQ << <_numBlocksM, _blockSize >> > (Q._matrixGPU, X._matrixGPU, Mu._matrixGPU, _rho, _sizeOPFTotal);
}



float OPFADMMGPU::updateRes(int indice)
{

	float resS = Y.max2(&Ypre);
	float resR = Y.max2(&X);
	float resV = 0;

	float oldrho = _rho;
	resF.set(0, indice, resR);
	resF.set(1, indice, oldrho * resS);
	resF.set(2, indice, resV);

	
	/*std::cout << " Y " << std::endl;
	Y.display(true);
	std::cout << " X " << std::endl;
	X.display(true);*/

	if (_tau > 1) {
		if (resR > _mu * resS) {
			_rho = _tau * _rho;
			Apt2.multiply(_tau);
			Hinv.divide(_tau);
		
			//std::cout << _iterGlobal << "rho augmente " << _rho << std::endl;
		}
		else if (resS > _mu * resR) {// rho = rho / tau_inc;
			_rho = _rho / _tau;
			Apt2.divide(_tau);

			Hinv.multiply(_tau);
			//std::cout << _tau << " " << _mu << std::endl;
			//std::cout << _iterGlobal << "rho diminue " << _rho << std::endl;
		}/**/
	}

	


	return MYMAX(MYMAX(resV, oldrho * resS), resR);
}

int OPFADMMGPU::feasiblePoint()
{
	bool mustTrans = false;
	if (X.getPos()) {
		X.transferCPU();
		_indiceBusBegin.transferCPU();
		mustTrans = true;
	}
	// X  (Pi, Qi, li, vi, pi, qi, vai, Pci ..., Qci... , lci...) !!!!!

	MatrixCPU test(_nBus, 1, -1);
	int counter = 0;
	for (int bus = 0; bus < _nBus; bus++) {
		int begin = _indiceBusBegin.get(bus, 0);
		float Si = X.get(begin, 0) * X.get(begin, 0) + X.get(begin + 1, 0) * X.get(begin + 1, 0);
		float li = X.get(begin + 2, 0);
		float vi = X.get(begin + 3, 0);
		float err = Si - li * vi;
		test.set(bus, 0, err);
		if (abs(err) > 0.0001) {
			counter++;
		}
	}
	//std::cout << " erreur sur la relaXation " << test.max2() << " " << counter << std::endl;
	//test.display();

	if (mustTrans) {
		X.transferGPU();
		_indiceBusBegin.transferGPU();
	}
	resF.set(2, (_iterGlobal - 1) / _stepG, test.max2());
	return counter;
}

MatrixCPU OPFADMMGPU::getPb(){
	bool transferToDo = false;
	if(Y.getPos()){
		Y.transferCPU();
		_indiceBusBegin.transferCPU();
		transferToDo = true;
	}
	MatrixCPU Pb(2*_nBus, 1);
	
	for (int i = 0; i <_nBus; i++)
	{
		Pb.set(i,0, Y.get(_indiceBusBegin.get(i, 0) + 4, 0));
		Pb.set(i + _nLine, 0, Y.get(_indiceBusBegin.get(i, 0) + 5, 0));
	}
	if(transferToDo){
		Y.transferGPU();
		_indiceBusBegin.transferGPU();
	}
	return Pb;
}
MatrixCPU OPFADMMGPU::getPhi(){
	bool transferToDo = false;
	if(Y.getPos()){
		Y.transferCPU();
		_indiceBusBegin.transferCPU();
		transferToDo = true;
	}
	MatrixCPU Phi(2*_nLine, 1);
	
	for (int i = 0; i <_nLine; i++)
	{
		Phi.set(i,0, Y.get(_indiceBusBegin.get(i + 1,0) + 0, 0));
		Phi.set(i + _nLine,0, Y.get(_indiceBusBegin.get(i + 1,0) + 1, 0));
	}
	if(transferToDo){
		Y.transferGPU();
		_indiceBusBegin.transferGPU();
	}
	return Phi;
	
}
MatrixCPU OPFADMMGPU::getE(){
	bool transferToDo = false;
	if(Y.getPos()){
		Y.transferCPU();
		_indiceBusBegin.transferCPU();
		transferToDo = true;
	}
	MatrixCPU E(2*_nBus, 1);
	
	for (int i = 0; i <_nBus; i++)
	{
		E.set(i,0, Y.get(_indiceBusBegin.get(i, 0) + 2, 0));
		E.set(i + _nLine,0, Y.get(_indiceBusBegin.get(i, 0) + 3, 0));
	}
	if(transferToDo){
		Y.transferGPU();
		_indiceBusBegin.transferGPU();
	}
	return E;
}



void OPFADMMGPU::display() {

	std::cout.precision(3);

	X.transferCPU();
	Y.transferCPU();
	Mu.transferCPU();
	Pn.transferCPU();
	_indiceBusBegin.transferCPU();
	_nAgentByBus.transferCPU();
	VoltageLimitReal.transferCPU();
	Pmin.transferCPU();
	Pmax.transferCPU();
	PnTmax.transferCPU();
	PnTmin.transferCPU();
	PnTilde.transferCPU();
	Cost1.transferCPU();
	Cost2.transferCPU();

	if (_iterGlobal == 0) {
		std::cout << "algorithm not launch" << std::endl;
	}
	else if (_iterGlobal < _iterG) {
		std::cout << "method " << _name << " converged in " << _iterGlobal << " iterations." << std::endl;
		std::cout << "Converged in " << (float)timeOPF / CLOCKS_PER_SEC << " seconds" << std::endl;

	}
	else {
		std::cout << "method " << _name << " not converged in " << _iterGlobal << " iterations." << std::endl;
		std::cout << "time taken " << (float)timeOPF / CLOCKS_PER_SEC << " seconds" << std::endl;
	}
	std::cout << "The power error of this state is (constraint) " << resF.get(0, _iterGlobal / _stepG) << " and convergence " << resF.get(1, _iterGlobal / _stepG) << std::endl;
	std::cout << "===============================================================|" << std::endl;
	std::cout << "      System Summary                                           |" << std::endl;
	std::cout << "===============================================================|" << std::endl;
	std::cout << "Buses            " << _nBus << std::endl;
	std::cout << "Branches         " << _nLine << std::endl;
	std::cout << "Agent            " << _nAgent << std::endl;
	std::cout << "Ploss            " << getPLoss() << std::endl;
	std::cout << "Qloss            " << getQLoss() << std::endl;


	std::cout << std::endl << std::endl;

	std::cout << "========================================================================================================|" << std::endl;
	std::cout << "      Bus Data                                                                                          |" << std::endl;
	std::cout << "========================================================================================================|" << std::endl;
	std::cout << " Bus |    Voltage  |   Power = Generation  + Load    |                Mu voltage and power              |" << std::endl;
	std::cout << "  #  |     Mag(pu) |    P (pu)      |      Q (pu)    |     V (pu)     |      P (pu)    |      Q (pu)    |" << std::endl;
	std::cout << "-----|-------------|----------------|----------------|----------------|----------------|----------------|" << std::endl;




	for (int b = 0; b < _nBus; b++) {
		int begining = _indiceBusBegin.get(b, 0);
		std::cout << std::setw(5) << b << "|" << std::setw(12) << sqrt(X.get(begining + 3, 0)) << " |" << std::setw(16)
			<< X.get(begining + 4, 0) << "|" << std::setw(16) << X.get(begining + 5, 0)
			<< "|" << std::setw(16) << Mu.get(begining + 3, 0) << "|" << std::setw(16)
			<< Mu.get(begining + 4, 0) << "|" << std::setw(16) << Mu.get(begining + 5, 0) << "|" << std::endl;

	}
	std::cout << std::endl << std::endl;
	std::cout << "========================================================================================================|" << std::endl;
	std::cout << "      Line Data                                                                                         |" << std::endl;
	std::cout << "========================================================================================================|" << std::endl;
	std::cout << " Line |    From     |    To      |                           Upstream flow                              |" << std::endl;
	std::cout << "  #   |    Bus      |    Bus     |    P (pu)      |    Q (pu)      |     l (pu)     |     Loss (pu)     |" << std::endl;
	std::cout << "------|-------------|------------|----------------|----------------|----------------|-------------------|" << std::endl;

	for (int l = 0; l < _nLine; l++) {
		int b = l + 1;
		int begining = _indiceBusBegin.get(b, 0);
		std::cout << std::setw(6) << l << "|" << std::setw(12) << CoresLineBus.get(l, 0) << " |" << std::setw(12)
			<< CoresLineBus.get(l, 1) << "|" << std::setw(16) << X.get(begining + 0, 0)
			<< "|" << std::setw(16) << X.get(begining + 1, 0) << "|" << std::setw(16)
			<< X.get(begining + 2, 0) << "|" << std::setw(19) << X.get(begining + 2, 0) * ZsRe.get(l, 0) << "|" << std::endl;
	}
	std::cout << std::endl << std::endl;
	std::cout << "========================================================================================================|" << std::endl;
	std::cout << "     Constraints                                                                                        |" << std::endl;
	std::cout << "========================================================================================================|" << std::endl;
	std::cout << " Bus | Voltage | Voltage | Voltage |        Power Injection          |          Power Injection         |" << std::endl;
	std::cout << "  #  | Mag(pu) | MIN(pu) |  MYMAX(pu)|  P (pu) | Pmin (pu) | Pmax (pu) |  Q (pu)  | Qmin (pu) | Qmax (pu) |" << std::endl;
	std::cout << "-----|---------|---------|---------|---------|-----------|-----------|----------|-----------|-----------|" << std::endl;


	for (int b = 0; b < _nBus; b++) {
		int begining = _indiceBusBegin.get(b, 0);
		int nb = _nAgentByBus.get(b, 0);
		std::cout << std::setw(5) << b << "|" << std::setw(8) << sqrt(Y.get(begining + 3, 0)) << " |" << std::setw(9)
			<< VoltageLimitReal.get(0, b) << "|" << std::setw(9) << VoltageLimitReal.get(1, b)
			<< "|" << std::setw(9) << Y.get(begining + 4, 0) << "|" << std::setw(11)
			<< PnTmin.get(b, 0) * nb << "|" << std::setw(11) << PnTmax.get(b, 0) * nb << "|" << std::setw(10) << Y.get(begining + 5, 0)
			<< "|" << std::setw(11) << PnTmin.get(b + _nBus, 0) * nb << "|" << std::setw(11) << PnTmax.get(b + _nBus, 0) * nb << "|" << std::endl;

	}
	std::cout << std::endl << std::endl;
	std::cout << "========================================================================================================|" << std::endl;
	std::cout << "      Agent Data                                                                                        |" << std::endl;
	std::cout << "========================================================================================================|" << std::endl;
	std::cout << " Agent |  Bus  |  Cost   |  Cost   |        Power Injection          |          Power Injection         |" << std::endl;
	std::cout << "  #    |   #   |  a (pu) |  b (pu) |  P (pu) | Pmin (pu) | Pmax (pu) |  Q (pu)  | Qmin (pu) | Qmax (pu) |" << std::endl;
	std::cout << "-------|-------|---------|---------|---------|-----------|-----------|----------|-----------|-----------|" << std::endl;

	for (int n = 0; n < _nAgent; n++) {
		int b = _CoresBusAgent.get(n, 0);
		std::cout << std::setw(7) << n << "|" << std::setw(7) << b << "|" << std::setw(8) << Cost1.get(n, 0) << " |" << std::setw(9)
			<< Cost2.get(n, 0) << "|" << std::setw(9) << Pn.get(n, 0) << "|" << std::setw(11)
			<< Pmin.get(n, 0) << "|" << std::setw(11) << Pmax.get(n, 0) << "|" << std::setw(10) << Pn.get(n + _nAgent, 0)
			<< "|" << std::setw(11) << Pmin.get(n + _nAgent, 0) << "|" << std::setw(11) << Pmax.get(n + _nAgent, 0) << "|" << std::endl;
	}


	std::cout << "========================================================================================================|" << std::endl;
	std::cout << "                      END PRINT                                                                         |" << std::endl;
	std::cout << "========================================================================================================|" << std::endl;

}




template <unsigned int _blockSizeSmall>
__global__ void updateChatGPU(float* Chat, float* Y, float* MU, float* nChild, float* Ancestor, float* posChild, float* Childs, float* indiceBusBegin, float* indiceChildBegin, float _rho, int nBus) {

	int bus = blockIdx.x;
	int index = threadIdx.x;
	int step = blockDim.x;

	__shared__ float shArr[_blockSizeSmall]; // c'est grand pour pas grand chose...


	int indice = indiceBusBegin[bus];
	int indiceChild = (bus < (nBus - 1)) ? indiceChildBegin[bus] : 0;
	int nb = nChild[bus];
	int Ai = Ancestor[bus];
	int c = posChild[bus];
	float var = 0;

	if (index < 6) {
		//float Phat, Qhat, lhat, phat, qhat;
		var = Y[indice + index] / 2 - MU[indice + index] / (2 * _rho);
		if (bus > 0) {
			if (index < 3) {
				int nAi = nChild[Ai];
				int indiceAncBus = indiceBusBegin[Ai] + 7 + nAi * index + c;
				//var = indiceAncBus;
				var += Y[indiceAncBus] / 2 - MU[indiceAncBus] / (2 * _rho);
			}
		}
	}
	float vhat = 0;
	float muhat = 0;
	for (int i = index; i < nb; i += step) {
		int Bus2 = Childs[indiceChild + i];
		int indiceBusChild = indiceBusBegin[Bus2];
		muhat += MU[indiceBusChild + 6]; // pas du tout coalescent
		vhat += Y[indiceBusChild + 6]; // pas du tout coalescent
	}
	shArr[index] = vhat / (nb + 1) - muhat / (_rho * (nb + 1));
	__syncthreads();
	for (int size = _blockSizeSmall / 2; size > 0; size /= 2) { //uniform
		if (index < size) {
			shArr[index] += shArr[index + size];
		}
		__syncthreads();
	}

	if (index < 6) {
		if (index == 3) {
			var = shArr[0] + Y[indice + 3] / (nb + 1) - MU[indice + 3] / (_rho * (nb + 1)); //shArr[0];
		}
		Chat[index * nBus + bus] = var; // pas coalescent mais bon perdu pour perdu
		// pour p et q, on a un /2 en trop !
	}
}

__global__ void updateBpt2(float* Bpt2, float* Chat, float* nAgentByBus, int nBus) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int step = blockDim.x * gridDim.x;


	for (int b = index; b < nBus; b += step) {
		int nA = nAgentByBus[b];

		Bpt2[b] = nA > 0 ? 2 * Chat[b + 4 * nBus] / nA : 0; // �criture coalescente et lecture coalescente
		Bpt2[b + nBus] = nA > 0 ? 2 * Chat[b + 5 * nBus] / nA : 0;

	}

}


template <unsigned int _blockSizeSmall>
__global__ void updatePnPGPUSharedResidual(float* Pn, float* PnPre, float* PnMoy, float* PnTilde, float* MUL, float* nAgentByBus, float _rhol, float* Ap2, float* Cp, float* Pmin,
	float* Pmax, float* Apt1, float* Apt2, float* Bpt2, float* CoresSoloBusAgent, float* CoresBusAgent, float* CoresBusAgentBegin, float eps, int nIterLMax, int nAgent, int nBus) {


	//Definition de toutes les variables locales
	int i = blockIdx.x; // c'est aussi l'identifiant du bus !
	unsigned int thIdx = threadIdx.x;

	// ne change pas

	float Ap2local[2];
	float Ap12local[2];
	float Cplocal[2];
	float Pminlocal[2];
	float Pmaxlocal[2];

	// constant et commun � tous les thread d'un bloc
	__shared__ float Apt1Shared;
	__shared__ float Apt2Shared;
	__shared__ float Apt12Shared;
	__shared__ float Bpt2Shared[2];
	__shared__ int nAgentShared;
	__shared__ float at1Shared;

	// change
	float Pnlocal[2];
	float Pnprelocal[2]; // change

	float bpt, MULOCAL, moy, p;
	float m, r, ub, lb, t;
	// le changement doit �tre partag� par tous les threads du bloc

	__shared__ float MuShared[2];
	__shared__ float PnMoyShared[2];
	__shared__ float PnTildeShared[2];
	__shared__ bool mustContinue;



	__shared__ float shArrP[_blockSizeSmall];
	__shared__ float shArrQ[_blockSizeSmall];


	if (thIdx == 0) {
		Apt1Shared = Apt1[i]; // rho_l *Ni, m�me pour les 2
		Apt2Shared = Apt2[i]; // rho * Ni^2, m�me pour les 2
		Apt12Shared = Apt1Shared + Apt2Shared; // m�me pour les 2
		nAgentShared = nAgentByBus[i];
		at1Shared = _rhol;
		mustContinue = false;
	}


	if (thIdx < 2) {
		Bpt2Shared[thIdx] = Bpt2[i + nBus * thIdx];
		MuShared[thIdx] = MUL[i + nBus * thIdx];
		PnMoyShared[thIdx] = PnMoy[i + nBus * thIdx];
		PnTildeShared[thIdx] = PnTilde[i + nBus * thIdx];
	}
	__syncthreads();

	int iter = 0;
	if (nAgentShared > 0) { // sinon il n'y a rien � faire
		const int CoresAgentLinLocal = CoresBusAgentBegin[i];
		const int j = CoresAgentLinLocal + thIdx;
		//const int endLocal = CoresAgentLinLocal + nAgentShared;
		double res = 0;
		if (nAgentShared == 1) { // cas trivial s'il n'y a qu'un agent, la divergence est entre les blocs donc c'est ok
			if (thIdx == 0) {
				int agent = CoresSoloBusAgent[i];
				// Cplocal et Ap12local, Pmaxlocal, Pminlocal � definir
				Cplocal[0] = Cp[agent];
				Ap2local[0] = Ap2[agent];
				ub = Pmax[agent];
				lb = Pmin[agent];
				r = (Apt2Shared * Bpt2Shared[0] - Cplocal[0]) / (Apt2Shared + Ap2local[0]); //pn = (_rho * Bpt2.get(b, 0) - Cost2.get(n, 0)) / ((_rho + Cost1.get(n, 0)));
				t = (ub - r) * (r > ub) + (lb - r) * (r < lb) + r;
				Pnlocal[0] = t;
				Pnprelocal[0] = t;
				PnMoyShared[0] = t;
				PnTildeShared[0] = t;

				// Q 
				Cplocal[1] = Cp[agent + nAgent];
				Ap2local[1] = Ap2[agent + nAgent];
				ub = Pmax[agent + nAgent];
				lb = Pmin[agent + nAgent];
				r = (Apt2Shared * Bpt2Shared[1] - Cplocal[1]) / (Apt2Shared + Ap2local[1]); //pn = (_rho * Bpt2.get(b, 0) - Cost2.get(n, 0)) / ((_rho + Cost1.get(n, 0)));
				t = (ub - r) * (r > ub) + (lb - r) * (r < lb) + r;
				Pnlocal[1] = t;
				Pnprelocal[1] = t;
				PnMoyShared[1] = t;
				PnTildeShared[1] = t;
			}
		}
		else {
			Pnlocal[0] = 0;
			Pnlocal[1] = 0;
			if (thIdx < nAgentShared)
			{
				int agent = CoresBusAgent[j];
				// P & Q
				Ap2local[0] = Ap2[agent];
				Ap12local[0] = Ap2local[0] + _rhol;
				Cplocal[0] = Cp[agent];
				Pminlocal[0] = Pmin[agent];
				Pmaxlocal[0] = Pmax[agent];
				Pnlocal[0] = Pn[agent];

				Ap2local[1] = Ap2[agent + nAgent];
				Ap12local[1] = Ap2local[1] + _rhol;
				Cplocal[1] = Cp[agent + nAgent];
				Pminlocal[1] = Pmin[agent + nAgent];
				Pmaxlocal[1] = Pmax[agent + nAgent];
				Pnlocal[1] = Pn[agent + nAgent];
			}

			//Calcul des it�rations

			for (iter = 0; iter < nIterLMax; iter++) {
				__syncthreads();
				if (thIdx < nAgentShared) {
					// P
					MULOCAL = MuShared[0];
					moy = PnMoyShared[0];
					p = PnTildeShared[0];

					Pnprelocal[0] = Pnlocal[0];
					m = Pnlocal[0] - moy + p - MULOCAL; // Pn.get(n, 0) - PnMoy.get(bus, 0) + PnTilde.get(bus, 0) - MuL.get(bus, 0);
					r = (m * at1Shared - Cplocal[0]) / Ap12local[0]; // pn = (Bp1.get(n, 0) * _rhol - Cost2.get(n, 0)) / Ap12.get(n, 0);
					ub = Pmaxlocal[0];
					lb = Pminlocal[0];
					t = (ub - r) * (r > ub) + (lb - r) * (r < lb) + r;
					Pnlocal[0] = t;
					
					res = (double) t - Pnprelocal[0];
					res = res * res;
					if (res > eps) {
						mustContinue = true; // pas de race condition, car l'ordre n'importe pas,
					}


					// Q
					MULOCAL = MuShared[1];
					moy = PnMoyShared[1];
					p = PnTildeShared[1];

					Pnprelocal[1] = Pnlocal[1];
					m = Pnlocal[1] - moy + p - MULOCAL; // Pn.get(n + _nAgent, 0) - PnMoy.get(bus + _nBus, 0) + PnTilde.get(bus + _nBus, 0) - MuL.get(bus + _nBus, 0);
					r = (m * at1Shared - Cplocal[1]) / Ap12local[1]; // pn = (Bp1.get(n + _nAgent, 0) * _rhol - Cost2.get(n + _nAgent, 0)) / Ap12.get(n+ _nAgent, 0);
					ub = Pmaxlocal[1];
					lb = Pminlocal[1];
					t = (ub - r) * (r > ub) + (lb - r) * (r < lb) + r;
					Pnlocal[1] = t;
					res = (double)t - Pnprelocal[1];
					res = res * res;
					if (res > eps) {
						mustContinue = true; // pas de race condition, car l'ordre n'importe pas,
					}

				}

				shArrP[thIdx] = Pnlocal[0];
				shArrQ[thIdx] = Pnlocal[1];


				__syncthreads();
				if (_blockSizeSmall >= 512) {
					if (thIdx < 256) {
						shArrP[thIdx] += shArrP[thIdx + 256];
						shArrQ[thIdx] += shArrQ[thIdx + 256];
					}
					__syncthreads();
				}
				if (_blockSizeSmall >= 256) {
					if (thIdx < 128) {
						shArrP[thIdx] += shArrP[thIdx + 128];
						shArrQ[thIdx] += shArrQ[thIdx + 128];
					}
					__syncthreads();
				}
				if (_blockSizeSmall >= 128) {
					if (thIdx < 64) {
						shArrP[thIdx] += shArrP[thIdx + 64];
						shArrQ[thIdx] += shArrQ[thIdx + 64];
					}
					__syncthreads();
				}
				if (_blockSizeSmall >= 64) {
					if (thIdx < 32) {
						warpReduce<_blockSizeSmall>(shArrP, thIdx);
						warpReduce<_blockSizeSmall>(shArrQ, thIdx);
					}
				}else if (_blockSizeSmall >= 32) {
					warpReduce<_blockSizeSmall>(shArrP, thIdx);
					warpReduce<_blockSizeSmall>(shArrQ, thIdx);
				}
				__syncthreads();

				if (thIdx == 0) {
					// P
					moy = shArrP[0] / nAgentShared;
					PnMoyShared[0] = moy;
					bpt = moy + MuShared[0]; //Bpt1.set(b, 0, MuL.get(b, 0) + PnMoy.get(b, 0));
					p = (Apt1Shared * bpt + Apt2Shared * Bpt2Shared[0]) / Apt12Shared; //pn = (Bpt1.get(b, 0) * Apt1.get(b, 0) + Bpt2.get(b, 0) * Apt2.get(b, 0)) / Apt12.get(b, 0);
					PnTildeShared[0] = p;
					res = p - moy;
					res = res * res;
					if (res > eps) {
						mustContinue = true;
					}
					MuShared[0] = MuShared[0] + moy - p; // mu = MuL.get(b, 0) + PnMoy.get(b, 0) - PnTilde.get(b, 0);
					
					// Q
					moy = shArrQ[0] / nAgentShared;
					PnMoyShared[1] = moy;
					bpt = moy + MuShared[1]; //Bpt1.set(b, 0, MuL.get(b, 0) + PnMoy.get(b, 0));
					p = (Apt1Shared * bpt + Apt2Shared * Bpt2Shared[1]) / Apt12Shared; //pn = (Bpt1.get(b + _nBus, 0) * Apt1.get(b + _nBus, 0) + Bpt2.get(b + _nBus, 0) * Apt2.get(b + _nBus, 0)) / Apt12.get(b + _nBus, 0);

					PnTildeShared[1] = p;
					res = p - moy;
					res = res * res;
					if (res > eps) {
						mustContinue = true;
					}
					MuShared[1] = MuShared[1] + moy - p; // mu = MuL.get(b, 0) + PnMoy.get(b, 0) - PnTilde.get(b, 0);

				}
				__syncthreads();
				if (!mustContinue) {
					break;
				}
				else {
					__syncthreads();
					if (thIdx == 0) {
						mustContinue = false;
					}
				}
			}
		}
		//Ecriture des it�rations
		__syncthreads();

		if (thIdx < nAgentShared)
		{
			int agent = CoresBusAgent[j];

			Pn[agent] = Pnlocal[0];
			PnPre[agent] = Pnprelocal[0];

			Pn[agent + nAgent] = Pnlocal[1];
			PnPre[agent + nAgent] = Pnprelocal[1];

		}
		if (thIdx == 0) {
			PnMoy[blockIdx.x] = PnMoyShared[0];// TMoyShared;
			PnTilde[blockIdx.x] = PnTildeShared[0];// PShared;
			MUL[blockIdx.x] = MuShared[0];// MuShared;

			PnMoy[blockIdx.x + nBus] = PnMoyShared[1];// TMoyShared;
			PnTilde[blockIdx.x + nBus] = PnTildeShared[1];// PShared;
			MUL[blockIdx.x + nBus] = MuShared[1];// MuShared;
		}
	}
}

