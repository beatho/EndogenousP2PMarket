#include "hip/hip_runtime.h"
	#include "../head/PACGPU.cuh"
#define MAX(X, Y) X * (X >= Y) + Y * (Y > X)
// betaLin, CoresAgentLinBig, indiceNu

PACGPU::PACGPU() : MethodP2P()
{
#if DEBUG_CONSTRUCTOR
	std::cout << " PACGPU Constructor" << std::endl;
#endif // DEBUG_CONSTRUCTOR
	_name = NAME;
}


PACGPU::PACGPU(float rho) : MethodP2P()
{
#if DEBUG_CONSTRUCTOR
	std::cout << "default PACGPU Constructor" << std::endl;
#endif // DEBUG_CONSTRUCTOR
	_name = NAME;
	_rho = rho;
}

PACGPU::~PACGPU()
{
	/*DELETEA(tempM1);
	DELETEA(tempM);
	DELETEA(X);
	DELETEA(Xpre);
	DELETEA(Xhat);
	DELETEA(Mu);
	DELETEA(Muhat);
	DELETEA(Nu);
	DELETEA(Nuhat);
	DELETEA(Hinv);
	DELETEA(H);
	DELETEA(matLb);
	DELETEA(Q);
	DELETEA(Qinit);
	DELETEA(matUb);*/
}
void PACGPU::setParam(float rho)
{
	_rho = rho;
}

void PACGPU::setGamma(float gamma)
{
	_gamma = gamma;
	if (_gammahat > _gamma) {
		std::cout << "Warning : gammahat should be smaller than gamma";
	}
}

void PACGPU::setGammahat(float gammahat)
{
	_gammahat = gammahat;
	if (_gammahat > _gamma) {
		std::cout << "Warning : gammahat should be smaller than gamma";
	}
}

void PACGPU::setInitCoef(float alpha, float phi, float theta)
{
	if (alpha <= 0 || phi < 0 || theta < 0 || alpha >= 1 || phi >= 1 || theta >= 1) {
		throw std::invalid_argument("coefficient must be positive and <1");
	}
	_alpha = alpha;
	_phi = phi;
	_theta = theta;
	
}

void PACGPU::setBestRhoGamma(float lambdaMax, float lambdaMin, const StudyCase& cas)
{
	throw std::runtime_error("setBestRhoGamma : WIP not implemented");
	/*MatrixGPU a = cas.geta();
	float alpha = 1 * a.min2();
	float L = 1 * a.max2();
	_gamma = (2 * alpha * L) / (2 * lambdaMax + lambdaMin);
	_gammahat = _gamma;
	if (augmente && _alpha>1) {
		_rho = 1 / (sqrt(_gamma * _alpha * lambdaMax));
	}
	else {
		_rho = 1 / (sqrt(_gamma * lambdaMax));
	}*/
	
	std::cout << "best gamma " << _gamma << " and rho " << _rho << std::endl;

}

void PACGPU::setBestRhoGammaHeuristic(const StudyCase& cas)
{
	int N = cas.getNagent();
	int Y = cas.getNCons();
	float lambdaMax = 1;
	float lambdaMin = 1;
	if (cas.isAC()) {
		lambdaMax = 0.9995 * N + 2.0455;
		lambdaMin = (1.9927 - 3.3289 / N) / N;
	}
	else {
		lambdaMax = abs(Y - 0.5 * N) + 0.4961 * N + 3.3561;
		if (N > 10) {
			lambdaMin = (-0.00675 * N + 0.9675) / (N * N) * (Y - 0.5 * N) * (Y - 0.5 * N) + (log(831 * N - 5755)) / (3 * N);
		}
		else {
			lambdaMin = 0.35 + 0.4676 / N;
		}
	}
	if (lambdaMin < 0) {
		lambdaMin = 0.000001;
	}

	MatrixCPU a = cas.geta();
	float alpha = 1 * a.min2Nnull(0.0001);
	float L = 1 * a.max2();
	//std::cout << "alpha " << alpha << " and L " << L << std::endl;
	_gamma = (2 * alpha * L) / (2 * lambdaMax + lambdaMin);
	_gammahat = _gamma;
	if (augmente && _alpha > 1) {
		_rho = 1 / (sqrt(_gamma * _alpha * lambdaMax));
	}
	else {
		_rho = 1 / (sqrt(_gamma * lambdaMax));
	}

	//std::cout << "best gamma " << _gamma << " and rho " << _rho << std::endl;
}

void PACGPU::updateCoef()
{
	throw std::runtime_error("updateCoef : WIP not implemented");
	/*if (augmente) {

		_rho = MAX(0.99 * _rho, 0.1);
		_rhoInv = 1 / _rho;
		for (int i = 0; i < _nAgent; i++) {
			H[i].set(0, 0, Cost1.get(i, 0) + _rhoInv);
			if (augmente) {
				H[i].increment(0, 0, _rho * _gamma);
			}
			int M = nVoisin.get(i, 0);
			for (int m = 0; m < M; m++) {

				H[i].set(m + 1, m + 1, _rhoInv); // diag tnm
				H[i].set(M + m + 1, M + m + 1, _rhoInv); // diag anm
				if (augmente) {
					H[i].set(m + 1, 0, -_rho * _gamma); // first column pn <-> tnm
					H[i].set(0, m + 1, -_rho * _gamma); // fisrt row    pn <-> tnm

					H[i].increment(m + 1, m + 1, 2 * _rho * _gamma); // diag tnm
					H[i].increment(M + m + 1, M + m + 1, 2 * _rho * _gamma); // diag anm
				}
			}
			for (int k = 1; k < M + 1; k++) {
				for (int j = 1; j < M + 1; j++) {
					if (k != j) {
						H[i].set(k, j, _rho * _gamma);
					}
				}
			}
			for (int k = 1; k < M + 1; k++) {
				for (int j = M + 1; j < 2 * M + 1; j++) {
					H[i].set(k, j, _rho * _gamma);
					H[i].set(j, k, _rho * _gamma);
				}
			}
			Hinv[i].invertEigen(&H[i]);
		}
	}
	*/
	
	/**/
	/*_alpha = 0.9 * _alpha;
	_theta = 0.9 * _theta;
	_phi = 0.9 * _phi;*/
}



void PACGPU::solve(Simparam* result, const Simparam& sim, const StudyCase& cas)
{
#ifdef DEBUG_SOLVE
	cas.display();
	sim.display(1);
#endif // DEBUG_SOLVE
	
	clock_t tall =clock();
#ifdef INSTRUMENTATION
	std::chrono::high_resolution_clock::time_point t1;
	std::chrono::high_resolution_clock::time_point t2;
#endif // INSTRUMENTATION

	// FB 0
	if (_id == 0) {
#ifdef INSTRUMENTATION
		t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION
		init(sim, cas);
#ifdef INSTRUMENTATION
		hipDeviceSynchronize();
		t2 = std::chrono::high_resolution_clock::now();
		timePerBlock.increment(0, 0, std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
		occurencePerBlock.increment(0, 0, 1);
#endif // INSTRUMENTATION
	}
	_rhog = sim.getRho();
	
	int iterG = sim.getIterG();
	int stepG = sim.getStepG();
	
	float epsG = sim.getEpsG()/2;
	
	
	float resG = 2 * epsG;
	int iterGlobal = 0;
	while ((iterGlobal < iterG) && (resG>epsG)) {

		updateLocalProb();
		
#ifdef INSTRUMENTATION
		occurencePerBlock.increment(0, 1, 1);
		//occurencePerBlock.increment(0, 2, 0);
		//occurencePerBlock.increment(0, 3, 0);

		t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION

		updateGlobalProb();

#ifdef INSTRUMENTATION
		hipDeviceSynchronize();
		t2 = std::chrono::high_resolution_clock::now();
		timePerBlock.increment(0, 5, std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
#endif // INSTRUMENTATION
		// FB 4
		if (!((iterGlobal -1) % stepG)) {
#ifdef INSTRUMENTATION
			t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION
			resG = updateRes(iterGlobal / stepG);
			//resG = 1;
#ifdef INSTRUMENTATION
			hipDeviceSynchronize();
			t2 = std::chrono::high_resolution_clock::now();
			timePerBlock.increment(0, 6, std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
#endif // INSTRUMENTATION
		}
		//std::cout << iterGlobal << " " << iterLocal << " " << resL << " " << resF.get(0, iterGlobal / stepG) << " " << resF.get(1, iterGlobal / stepG) << std::endl;

		iterGlobal++;
	}
	//std::cout << iterGlobal  << " " << resF.get(0, (iterGlobal - 1) / stepG) << " " << resF.get(1, (iterGlobal - 1) / stepG) << " " << resF.get(2, (iterGlobal - 1) / stepG) << std::endl;


	
	
#ifdef INSTRUMENTATION	
	occurencePerBlock.increment(0, 5, iterGlobal);
	occurencePerBlock.increment(0, 6, iterGlobal / stepG);
	

	t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION
	MatrixCPU tradeLinCPU;
	X.toMatCPU(tradeLinCPU);
	MatrixCPU CoresAgentLinCPU;
	CoresAgentLin.toMatCPU(CoresAgentLinCPU);
	MatrixCPU CoresLinVoisinCPU;
	CoresLinVoisin.toMatCPU(CoresLinVoisinCPU);

	//MatrixCPU MUCPU;
	//MU.toMatCPU(MUCPU);
	
	
	for (int idAgent = 0; idAgent < _nAgentTrue; idAgent++) {
		int begin = CoresAgentLinCPU.get(idAgent, 0);
		int Nvoisinmax = nVoisinCPU.get(idAgent, 0);
		for (int voisin = 0; voisin < Nvoisinmax; voisin++) {
			int idVoisin = CoresLinVoisinCPU.get(begin + voisin + 1,0);
			trade.set(idAgent, idVoisin, tradeLinCPU.get(begin + voisin + 1, 0));
		}
		Pn.set(idAgent, 0, tradeLinCPU.get(begin, 0));
	}
	int indice = 0;
	for (int idAgent = _nAgentTrue; idAgent < _nAgent; idAgent++) {
		indice = 0;
		int begin = CoresAgentLinCPU.get(idAgent, 0);
		for (int idVoisin = 0; idVoisin < _nAgentTrue; idVoisin++) {
			if (idVoisin != (idAgent - _nAgentTrue)) {
				trade.set(idAgent, idVoisin, tradeLinCPU.get(begin + indice + 1, 0));
				indice = indice + 1;
			}
			Pn.set(idAgent, 0, tradeLinCPU.get(begin, 0));
		}
	}
	//std::cout << "Pn :" << std::endl;
	//Pn.display();
	
	
	calculFcPAC << <_nAgent, _blockSize >> > (tempN1._matrixGPU, tempNN._matrixGPU, Cost1._matrixGPU, Cost2._matrixGPU, Qinit._matrixGPU, X._matrixGPU, CoresAgentLin._matrixGPU, nVoisin._matrixGPU);
	
	float fc = tempN1.sum();

	fc += tempNN.sum();
	
	 //fc = calcFc(&Cost1, &Cost2, &trade, &Pn, &BETA, &tempN1, &tempNN);
	// FB 5
	
	//std::cout << "set end" << std::endl;


	result->setResF(&resF);
	result->setTrade(&trade); 
	result->setIter(iterGlobal);
	result->setPn(&Pn);
	result->setFc(fc);
	
#ifdef INSTRUMENTATION
	hipDeviceSynchronize();
	t2 = std::chrono::high_resolution_clock::now();
	timePerBlock.increment(0, 7, std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
	occurencePerBlock.increment(0, 7, 1);

	result->setTimeBloc(&timePerBlock, &occurencePerBlock);
#endif // INSTRUMENTATION
	tall = clock() - tall;

	result->setTime((float)tall / CLOCKS_PER_SEC);
	
}

void PACGPU::updateP0(const StudyCase& cas)
{
	_id = _id + 1;
#ifdef INSTRUMENTATION
	std::chrono::high_resolution_clock::time_point t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION

	

	MatrixGPU Pmin(cas.getPmin(), 1);
	MatrixGPU Pmax(cas.getPmax(), 1);
	MatrixGPU Cost2(cas.getb(), 1);
	MatrixGPU Lb(cas.getLb(), 1);
	MatrixGPU Ub(cas.getUb(), 1);

	updateP0PAC << <_nAgent, _blockSize >> > (matLb._matrixGPU, matUb._matrixGPU, Q._matrixGPU, Qinit._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, Cost2._matrixGPU, Lb._matrixGPU, Ub._matrixGPU, CoresAgentLin._matrixGPU, nVoisin._matrixGPU);
	
#ifdef INSTRUMENTATION
	hipDeviceSynchronize();
	std::chrono::high_resolution_clock::time_point t2 = std::chrono::high_resolution_clock::now();
	timePerBlock.increment(0, 8, std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
	occurencePerBlock.increment(0, 8, 1);
#endif // INSTRUMENTATION

}

void PACGPU::init(const Simparam& sim, const StudyCase& cas)
{
	// initialisation des matrixs et variables 
	
	clock_t t = clock();
	//std::cout << "init " << std::endl;
	if (_rho == 0) {
		_rho = sim.getRho();
	}

	_rhog = _rho;
	const int iterG = sim.getIterG();
	const int stepG = sim.getStepG();
	float epsG = sim.getEpsG();
	isAC = cas.isAC();
	
	_nAgentTrue = sim.getNAgent();
	_nAgent = _nAgentTrue + isAC * _nAgentTrue;
	_numBlocksN = ceil((_nAgent + _blockSize - 1) / _blockSize);
	nVoisin = MatrixGPU(cas.getNvoi(), 1);
	nVoisin.preallocateReduction();
	nVoisinCPU = cas.getNvoi();


	_nTrade = nVoisin.sum();
	_nTradeP = 0;
	if (isAC) {
		for (int n = 0; n < _nAgentTrue; n++) {
			_nTradeP += nVoisinCPU.get(n, 0);
		}
		_nTradeQ = _nTrade - _nTradeP;
		if (_nTradeQ != (_nAgentTrue * (_nAgentTrue - 1))) {
			std::cout << "err PACGPU : " << _nAgent << " " << _nAgentTrue << " " << _nTrade << " " << _nTradeP << " " << _nTradeQ << std::endl;

			throw std::invalid_argument("Agent must be fully connected for the Q echanges, WIP");
		}
	}
	else {
		_nTradeP = _nTrade;
	}
	//_sizePACGPU = _nAgent + 2 * _nTrade;


	_rhoInv = 1 / _rhog;
	resF = MatrixCPU(3, (iterG / stepG) + 1);

	

	_sizePACX = _nAgent + 2 * _nTrade; //_nAgent * (1 + 2*Mn)
	_sizePACMu = _nAgent + _nTrade; // _nAgent * (1 + Mn)
	_sizePACNu = _nTrade; // _nAgent * Mn
	_sizeHinv = 1 + 2 * nVoisin.max2();


	if (CoresMatLin.getPos()) { // si sur GPU il faut refaire les transferts
		CoresIndiceNu.transferCPU();
		CoresAgentLin.transferCPU();
		CoresAgentLinBig.transferCPU();
		_sizeQ.transferCPU();
		CoresLinAgent.transferCPU();
		CoresLinVoisin.transferCPU();
		CoresLinTrans.transferCPU();
		CoresMatLin.transferCPU();
		CoresLinTransLocal.transferCPU();

		X.transferCPU();
		matLb.transferCPU();
		matUb.transferCPU();
		Hinv.transferCPU();
		Qinit.transferCPU();
		Cost1.transferCPU();
		Cost2.transferCPU();
	}
	BETA = MatrixGPU(cas.getBeta());
	MatrixCPU Ub(cas.getUb());
	MatrixCPU Lb(cas.getLb());
	MatrixCPU Pmin(cas.getPmin());
	MatrixCPU Pmax(cas.getPmax());
	Cost1 = cas.geta();
	Cost2 = cas.getb();

	trade = sim.getTrade();
	Pn = sim.getPn();

	CoresMatLin = MatrixGPU(_nAgent, _nAgentTrue, -1);
	CoresLinAgent = MatrixGPU(_sizePACX, 1, -1);
	CoresAgentLin = MatrixGPU(_nAgent, 1, -1);
	CoresAgentLinBig = MatrixGPU(_sizePACX, 1, -1);
	CoresLinVoisin = MatrixGPU(_sizePACX, 1, -1);
	CoresLinTrans = MatrixGPU(_sizePACX, 1, -1);
	CoresLinTransLocal = MatrixGPU(_sizePACX, 1, -1);
	CoresIndiceNu = MatrixGPU(_sizePACX, 1, -1);
	_sizeQ = MatrixGPU(_sizePACX, 1, -1);
	
	// Who is the peer ? 
	int indice = 0;
	int debutNu = 0;
	
	//std::cout << " P " << std::endl;
	for (int idAgent = 0; idAgent < _nAgentTrue; idAgent++) { // P
		MatrixGPU omega(cas.getVoisin(idAgent));
		int Nvoisinmax = nVoisinCPU.get(idAgent, 0);
		int begin = indice;
		
		
		CoresIndiceNu.set(indice, 0, debutNu); // le debut de Mu[agent]
		CoresAgentLin.set(idAgent, 0, indice);

		// Pn
		CoresAgentLinBig.set(indice, 0, begin);
		_sizeQ.set(indice, 0, 1 + 2 * Nvoisinmax);
		indice += 1; 
		debutNu += Nvoisinmax;
		for (int voisin = 0; voisin < Nvoisinmax; voisin++) {
			int idVoisin = omega.get(voisin, 0);
			CoresAgentLinBig.set(indice, 0, begin);
			CoresLinAgent.set(indice, 0, idAgent);
			CoresLinVoisin.set(indice, 0, idVoisin); // 
			CoresAgentLinBig.set(indice + Nvoisinmax, 0, begin);
			CoresLinAgent.set(indice + Nvoisinmax, 0, idVoisin);
			CoresLinVoisin.set(indice + Nvoisinmax, 0, idAgent); // 
			_sizeQ.set(indice, 0, 1 + 2 * Nvoisinmax);
			_sizeQ.set(indice + Nvoisinmax, 0, 1 + 2 * Nvoisinmax);
			CoresMatLin.set(idAgent, idVoisin, indice);
			indice = indice + 1;
		}
		indice += Nvoisinmax;
	}
	//std::cout << " Q " << std::endl;
	for (int idAgent = _nAgentTrue; idAgent < _nAgent; idAgent++) { // Q
		int begin = indice;
		int Nvoisinmax = (_nAgentTrue - 1);
		
		CoresIndiceNu.set(indice, 0, debutNu); // le debut de Mu[agent]
		CoresAgentLin.set(idAgent, 0, indice);

		CoresAgentLinBig.set(indice, 0, begin);
		_sizeQ.set(indice, 0, 1 + 2 * Nvoisinmax);
		indice += 1;
		debutNu += Nvoisinmax;
		for (int idVoisin = 0; idVoisin < _nAgentTrue; idVoisin++) {
			if (idVoisin != (idAgent - _nAgentTrue)) {
				CoresAgentLinBig.set(indice, 0, begin);
				CoresAgentLinBig.set(indice + Nvoisinmax, 0, begin);
				CoresLinAgent.set(indice, 0, idAgent);
				CoresLinVoisin.set(indice, 0, idVoisin + _nAgentTrue);
				CoresLinAgent.set(indice + Nvoisinmax, 0, idVoisin + _nAgentTrue);
				CoresLinVoisin.set(indice + Nvoisinmax, 0, idAgent);
				CoresMatLin.set(idAgent, idVoisin, indice);
				_sizeQ.set(indice, 0, 1 + 2 * Nvoisinmax);
				_sizeQ.set(indice + Nvoisinmax, 0, 1 + 2 * Nvoisinmax);
				indice = indice + 1;
			}
		}
		indice += Nvoisinmax;
	}
	
	/*CoresLinAgent.display();
	CoresLinVoisin.display();*/

	for (int idAgent = 0; idAgent < _nAgent; idAgent++) {
		int debut = CoresAgentLin.get(idAgent, 0, 0);
		int Nvoisinmax = nVoisinCPU.get(idAgent, 0);

		for (int m = 0; m < Nvoisinmax; m++) {
			int lin = debut + m + 1;
			int voisin = CoresLinVoisin.get(lin, 0);
			int k = 0;
			if (idAgent < _nAgentTrue) // P
				k = CoresMatLin.get(voisin, idAgent);
			else {
				k = CoresMatLin.get(voisin, idAgent - _nAgentTrue);
			}

			//std::cout << " trade num " << lin << " entre " << idAgent << " et " << voisin;
			//std::cout << " le trade symetrique le trade num " << k << std::endl;
			CoresLinTrans.set(lin, 0, k);
		}
	}

	for (int i = 0; i < _nAgent; i++) {
		int M = nVoisinCPU.get(i, 0);
		for (int m = 0; m < M; m++) {
			int lin = CoresAgentLin.get(i, 0) + m + 1; // indice global de tim = tip (m est le num�ro du voisin, p est le num�ro de l'agent)
			int p = CoresLinVoisin.get(lin, 0); // valeur de p
			int lin2 = CoresLinTrans.get(lin, 0); // indice global de tpi
			int linLoc = lin2 - CoresAgentLin.get(p, 0) - 1; //indice local de tpi
			CoresLinTransLocal.set(lin, 0, linLoc);

			int debutNuTrans = CoresIndiceNu.get(CoresAgentLin.get(p, 0), 0);
			CoresIndiceNu.set(lin, 0, debutNuTrans + linLoc);

			//std::cout << lin << ": trade entre " << i << " et " << p << " num local " << m ;
			//std::cout << " le symetrique est " << lin2 <<  " ou " << linLoc << std::endl;
		}
	}
	//CoresAgentLin.display();
	//CoresLinTrans.display();
	//CoresIndiceNu.display();
	

	//std::cout << " creation" << std::endl;
	
	X = MatrixGPU(_sizePACX, 1);
	Xpre = MatrixGPU(_sizePACX, 1, 0, 1);
	Xhat = MatrixGPU(_sizePACX, 1, 0, 1);
	matLb = MatrixGPU(_sizePACX, 1);
	matUb = MatrixGPU(_sizePACX, 1);

	Mu = MatrixGPU(_sizePACMu, 1, 0, 1);
	Muhat = MatrixGPU(_sizePACMu, 1, 0, 1);
	Nu = MatrixGPU(_sizePACNu, 1, 0, 1);
	Nuhat = MatrixGPU(_sizePACNu, 1, 0, 1);

	tempN1 = MatrixGPU(_nAgent, 1, 0, 1);
	tempNN = MatrixGPU(_nAgent, _nAgentTrue, 0, 1);
	tempM1 = MatrixGPU(_sizePACMu, 1, 0, 1);
	tempM = MatrixGPU(_sizePACNu, 1, 0, 1);


	Hinv = MatrixGPU(_sizePACX, _sizeHinv);
	Q = MatrixGPU(_sizePACX, 1, 0, 1);
	Qinit = MatrixGPU(_sizePACX, 1);
	
	//std::cout << _sizePACX << " " << _sizeHinv << std::endl;

	//std::cout << "problem setup " << std::endl;
	for (int i = 0; i < _nAgent; i++) {
		//std::cout << "********* Agent " << i<< "**********"<< std::endl;
		// def
		int M = nVoisinCPU.get(i, 0);
		int begin = CoresAgentLin.get(i, 0);
		MatrixCPU Htemp(1 + 2 * M, 1 + 2 * M);
		H = MatrixCPU(1 + 2 * M, 1 + 2 * M);
		
		
		// init
		X.set(begin, 0, Pn.get(i, 0));
		matLb.set(begin, 0, Pmin.get(i, 0));
		matUb.set(begin, 0, Pmax.get(i, 0));
		H.set(0, 0, Cost1.get(i, 0) + _rhoInv);
		if (augmente) {
			H.increment(0, 0, _rho * _gamma);
		}
		
		Qinit.set(begin, 0, Cost2.get(i, 0));
		
		for (int m = 0; m < M; m++) {
			int indice = begin + m + 1;
			int voisin = CoresLinVoisin.get(indice, 0);
			_sizeQ.set(indice, 0, 1 + 2 * M);
			_sizeQ.set(indice + M, 0, 1 + 2 * M);
			//std::cout << "voisin num " << m << " is " << voisin << std::endl;

			X.set(indice, 0, trade.get(i, voisin % _nAgentTrue)); // tnm
			X.set(indice + M , 0, trade.get(voisin, i % _nAgentTrue)); //amn
			matLb.set(indice, 0, Lb.get(i, 0));
			matLb.set(indice + M, 0, -Ub.get(i, 0)); // est ce que cela g�ne la convergence ou est ce que cela l'aide ?
			matUb.set(indice, 0, Ub.get(i, 0));
			matUb.set(indice + M, 0, -Lb.get(i, 0));
			
			H.set(m + 1, m + 1, _rhoInv); // diag tnm
			H.set(M + m + 1, M + m + 1, _rhoInv); // diag anm
			if (augmente) {
				H.set(m + 1, 0, -_rho * _gamma); // first column pn <-> tnm
				H.set(0, m + 1, -_rho * _gamma); // fisrt row    pn <-> tnm

				H.increment(m + 1, m + 1, 2 * _rho * _gamma); // diag tnm
				H.increment(M + m + 1, M + m + 1, 2 * _rho * _gamma); // diag anm
			}
			if (i < _nAgentTrue) {
				Qinit.set(indice, 0, BETA.get(i, voisin));
			}
		}
		
		if (augmente) {
			for (int k = 1; k < M + 1; k++) {
				for (int j = 1; j < M + 1; j++) {
					if (k != j) {
						H.set(k, j, _rho * _gamma); // sum(sum(tnk*tnj)) -> ||Ga||
					}
				}
			}
			for (int k = 1; k < M + 1; k++) {
				int j = M + k;
				H.set(k, j, _rho * _gamma);
				H.set(j, k, _rho * _gamma);
			}
		}	
		//H.display(); 
		
		Htemp.invertGaussJordan(&H);
		
		
		
		Hinv.setBloc(begin, begin + 2 * M + 1, 0, 1 + 2 * M, &Htemp);
		
		
		
	}
	//Hinv.display();
	//std::cout << std::endl;
	//CoresAgentLinBig.display();
	//std::cout << " Transfert GPU " <<std::endl;
	
	//CoresAgentLin.display();
	//CoresLinTrans.display();



	CoresMatLin.transferGPU();
	CoresLinAgent.transferGPU();
	CoresAgentLin.transferGPU();
	CoresAgentLinBig.transferGPU();
	CoresLinVoisin.transferGPU();
	CoresLinTrans.transferGPU();
	CoresLinTransLocal.transferGPU();
	CoresIndiceNu.transferGPU();
	
	_sizeQ.transferGPU();
	X.transferGPU();

	X.preallocateReduction();
	Muhat.preallocateReduction();
	Nuhat.preallocateReduction();
	tempM1.preallocateReduction();
	tempM.preallocateReduction();
	tempN1.preallocateReduction();
	tempNN.preallocateReduction();

	matLb.transferGPU();
	matUb.transferGPU();
	Hinv.transferGPU();
	
	Qinit.transferGPU();

	Xpre.set(&X);

	Cost1.transferGPU();
	Cost2.transferGPU();
	


	//matLb.display(true);
	//matUb.display(true);

	//std::cout << "Global update" << std::endl;
	
	
	updateGlobalProb();
	


	//std::cout << "rho " << _rhog << " _alpha " << _alpha << " _phi " << _phi << std::endl;
	//std::cout << "fin init temps : " << (float)(clock() - t) / CLOCKS_PER_SEC << std::endl;
}

void PACGPU::setBestParam(const StudyCase& cas)
{
	setBestRhoGammaHeuristic(cas);
}


void PACGPU::updateGlobalProb() {

	// communication of xhat
	updateNu();
	//Nu.display(true);
	//Nuhat.display(true);
	// communication of nuhat
	updateQ();
	if (augmente) {
		//updateCoef();
	}
	//Q.display(true);
	//std::cout << "******" << std::endl;
}

void PACGPU::updateLocalProb() {
	// FB 1a
#ifdef INSTRUMENTATION
	std::chrono::high_resolution_clock::time_point t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION
		
	
	Xpre.swap(&X);
	int numBlock = _sizePACX;
	switch (_blockSize) {
	case 512:
		updateLocalProblPAC<512> <<<numBlock, _blockSize >> > (X._matrixGPU, Q._matrixGPU, Hinv._matrixGPU, matLb._matrixGPU, matUb._matrixGPU, CoresAgentLinBig._matrixGPU, _sizeQ._matrixGPU, _sizeHinv);
		break;
	case 256:
		updateLocalProblPAC<256> << <numBlock, _blockSize >> > (X._matrixGPU, Q._matrixGPU, Hinv._matrixGPU, matLb._matrixGPU, matUb._matrixGPU, CoresAgentLinBig._matrixGPU, _sizeQ._matrixGPU, _sizeHinv);
		break;
	case 128:
		updateLocalProblPAC<128> << <numBlock, _blockSize >> > (X._matrixGPU, Q._matrixGPU, Hinv._matrixGPU, matLb._matrixGPU, matUb._matrixGPU, CoresAgentLinBig._matrixGPU, _sizeQ._matrixGPU, _sizeHinv);
		break;
	case 64:
		updateLocalProblPAC< 64> << <numBlock, _blockSize >> > (X._matrixGPU, Q._matrixGPU, Hinv._matrixGPU, matLb._matrixGPU, matUb._matrixGPU, CoresAgentLinBig._matrixGPU, _sizeQ._matrixGPU, _sizeHinv);
		break;
	case 32:
		updateLocalProblPAC< 32> << <numBlock, _blockSize >> > (X._matrixGPU, Q._matrixGPU, Hinv._matrixGPU, matLb._matrixGPU, matUb._matrixGPU, CoresAgentLinBig._matrixGPU, _sizeQ._matrixGPU, _sizeHinv);
		break;
	case 16:
		updateLocalProblPAC< 16> << <numBlock, _blockSize >> > (X._matrixGPU, Q._matrixGPU, Hinv._matrixGPU, matLb._matrixGPU, matUb._matrixGPU, CoresAgentLinBig._matrixGPU, _sizeQ._matrixGPU, _sizeHinv);
		break;
	case  8:
		updateLocalProblPAC<  8> << <numBlock, _blockSize >> > (X._matrixGPU, Q._matrixGPU, Hinv._matrixGPU, matLb._matrixGPU, matUb._matrixGPU, CoresAgentLinBig._matrixGPU, _sizeQ._matrixGPU, _sizeHinv);
		break;
	case  4:
		updateLocalProblPAC<  4> << <numBlock, _blockSize >> > (X._matrixGPU, Q._matrixGPU, Hinv._matrixGPU, matLb._matrixGPU, matUb._matrixGPU, CoresAgentLinBig._matrixGPU, _sizeQ._matrixGPU, _sizeHinv);
		break;
	case  2:
		updateLocalProblPAC<  2> << <numBlock, _blockSize >> > (X._matrixGPU, Q._matrixGPU, Hinv._matrixGPU, matLb._matrixGPU, matUb._matrixGPU, CoresAgentLinBig._matrixGPU, _sizeQ._matrixGPU, _sizeHinv);
		break;
	case  1:
		updateLocalProblPAC<  1> << <numBlock, _blockSize >> > (X._matrixGPU, Q._matrixGPU, Hinv._matrixGPU, matLb._matrixGPU, matUb._matrixGPU, CoresAgentLinBig._matrixGPU, _sizeQ._matrixGPU, _sizeHinv);
		break;
	}
	
	//X.display(true);
	updateXhat();
	//Xhat.display(true);
	updateMu();
	//Mu.display(true);
	//Muhat.display(true);
#ifdef INSTRUMENTATION
	hipDeviceSynchronize();
	std::chrono::high_resolution_clock::time_point t2 = std::chrono::high_resolution_clock::now();
	timePerBlock.increment(0, 1, std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
#endif // INSTRUMENTATION	

	

}

void PACGPU::updateXhat()
{
	
	if (augmente) {
		//Xhat.subtract(&X, &Xhat);
		Xhat.subtract(&X, &Xpre);
		Xhat.multiply(_alpha);
		Xhat.add(&X);
	}
	else {
		Xhat.set(&X);
	}

}

void PACGPU::updateMu()
{
	int numBlock = _nAgent;

	if (augmente) {
		switch (_blockSize) {
		case 512:
			updatePACMuAug<512> << <numBlock, _blockSize >> > (Mu._matrixGPU, Muhat._matrixGPU, Xhat._matrixGPU, nVoisin._matrixGPU, CoresAgentLin._matrixGPU, CoresIndiceNu._matrixGPU, _rho, _gamma, _phi);
			break;
		case 256:
			updatePACMuAug<256> << <numBlock, _blockSize >> > (Mu._matrixGPU, Muhat._matrixGPU, Xhat._matrixGPU, nVoisin._matrixGPU, CoresAgentLin._matrixGPU, CoresIndiceNu._matrixGPU, _rho, _gamma, _phi);
			break;
		case 128:
			updatePACMuAug<128> << <numBlock, _blockSize >> > (Mu._matrixGPU, Muhat._matrixGPU, Xhat._matrixGPU, nVoisin._matrixGPU, CoresAgentLin._matrixGPU, CoresIndiceNu._matrixGPU, _rho, _gamma, _phi);
			break;
		case 64:
			updatePACMuAug< 64> << <numBlock, _blockSize >> > (Mu._matrixGPU, Muhat._matrixGPU, Xhat._matrixGPU, nVoisin._matrixGPU, CoresAgentLin._matrixGPU, CoresIndiceNu._matrixGPU, _rho, _gamma, _phi);
			break;
		case 32:
			updatePACMuAug< 32> << <numBlock, _blockSize >> > (Mu._matrixGPU, Muhat._matrixGPU, Xhat._matrixGPU, nVoisin._matrixGPU, CoresAgentLin._matrixGPU, CoresIndiceNu._matrixGPU, _rho, _gamma, _phi);
			break;
		case 16:
			updatePACMuAug< 16> << <numBlock, _blockSize >> > (Mu._matrixGPU, Muhat._matrixGPU, Xhat._matrixGPU, nVoisin._matrixGPU, CoresAgentLin._matrixGPU, CoresIndiceNu._matrixGPU, _rho, _gamma, _phi);
			break;
		case  8:
			updatePACMuAug<  8> << <numBlock, _blockSize >> > (Mu._matrixGPU, Muhat._matrixGPU, Xhat._matrixGPU, nVoisin._matrixGPU, CoresAgentLin._matrixGPU, CoresIndiceNu._matrixGPU, _rho, _gamma, _phi);
			break;
		case  4:
			updatePACMuAug<  4> << <numBlock, _blockSize >> > (Mu._matrixGPU, Muhat._matrixGPU, Xhat._matrixGPU, nVoisin._matrixGPU, CoresAgentLin._matrixGPU, CoresIndiceNu._matrixGPU, _rho, _gamma, _phi);
			break;
		case  2:
			updatePACMuAug<  2> << <numBlock, _blockSize >> > (Mu._matrixGPU, Muhat._matrixGPU, Xhat._matrixGPU, nVoisin._matrixGPU, CoresAgentLin._matrixGPU, CoresIndiceNu._matrixGPU, _rho, _gamma, _phi);
			break;
		case  1:
			updatePACMuAug<  1> << <numBlock, _blockSize >> > (Mu._matrixGPU, Muhat._matrixGPU, Xhat._matrixGPU, nVoisin._matrixGPU, CoresAgentLin._matrixGPU, CoresIndiceNu._matrixGPU, _rho, _gamma, _phi);
			break;
		}
	}
	else {
		switch (_blockSize) {
		case 512:
			updatePACMu<512> << <numBlock, _blockSize >> > (Mu._matrixGPU, Muhat._matrixGPU, Xhat._matrixGPU, nVoisin._matrixGPU, CoresAgentLin._matrixGPU, CoresIndiceNu._matrixGPU, _rho, _gamma, _gammahat);
			break;
		case 256:
			updatePACMu<256> << <numBlock, _blockSize >> > (Mu._matrixGPU, Muhat._matrixGPU, Xhat._matrixGPU, nVoisin._matrixGPU, CoresAgentLin._matrixGPU, CoresIndiceNu._matrixGPU, _rho, _gamma, _gammahat);
			break;
		case 128:
			updatePACMu<128> << <numBlock, _blockSize >> > (Mu._matrixGPU, Muhat._matrixGPU, Xhat._matrixGPU, nVoisin._matrixGPU, CoresAgentLin._matrixGPU, CoresIndiceNu._matrixGPU, _rho, _gamma, _gammahat);
			break;
		case 64:
			updatePACMu< 64> << <numBlock, _blockSize >> > (Mu._matrixGPU, Muhat._matrixGPU, Xhat._matrixGPU, nVoisin._matrixGPU, CoresAgentLin._matrixGPU, CoresIndiceNu._matrixGPU, _rho, _gamma, _gammahat);
			break;
		case 32:
			updatePACMu< 32> << <numBlock, _blockSize >> > (Mu._matrixGPU, Muhat._matrixGPU, Xhat._matrixGPU, nVoisin._matrixGPU, CoresAgentLin._matrixGPU, CoresIndiceNu._matrixGPU, _rho, _gamma, _gammahat);
			break;
		case 16:
			updatePACMu< 16> << <numBlock, _blockSize >> > (Mu._matrixGPU, Muhat._matrixGPU, Xhat._matrixGPU, nVoisin._matrixGPU, CoresAgentLin._matrixGPU, CoresIndiceNu._matrixGPU, _rho, _gamma, _gammahat);
			break;
		case  8:
			updatePACMu<  8> << <numBlock, _blockSize >> > (Mu._matrixGPU, Muhat._matrixGPU, Xhat._matrixGPU, nVoisin._matrixGPU, CoresAgentLin._matrixGPU, CoresIndiceNu._matrixGPU, _rho, _gamma, _gammahat);
			break;
		case  4:
			updatePACMu<  4> << <numBlock, _blockSize >> > (Mu._matrixGPU, Muhat._matrixGPU, Xhat._matrixGPU, nVoisin._matrixGPU, CoresAgentLin._matrixGPU, CoresIndiceNu._matrixGPU, _rho, _gamma, _gammahat);
			break;
		case  2:
			updatePACMu<  2> << <numBlock, _blockSize >> > (Mu._matrixGPU, Muhat._matrixGPU, Xhat._matrixGPU, nVoisin._matrixGPU, CoresAgentLin._matrixGPU, CoresIndiceNu._matrixGPU, _rho, _gamma, _gammahat);
			break;
		case  1:
			updatePACMu<  1> << <numBlock, _blockSize >> > (Mu._matrixGPU, Muhat._matrixGPU, Xhat._matrixGPU, nVoisin._matrixGPU, CoresAgentLin._matrixGPU, CoresIndiceNu._matrixGPU, _rho, _gamma, _gammahat);
			break;
		}
	}

	
}



void PACGPU::updateNu()
{
	if (augmente) {
		updateNuAug <<<_nAgent, _blockSize >> > (Nu._matrixGPU, Nuhat._matrixGPU, Xhat._matrixGPU, nVoisin._matrixGPU, CoresAgentLin._matrixGPU, CoresLinTrans._matrixGPU, CoresIndiceNu._matrixGPU, _rho, _gamma, _theta);
	}
	else {
		updateNuGPU << <_nAgent, _blockSize >> > (Nu._matrixGPU, Nuhat._matrixGPU, Xhat._matrixGPU, nVoisin._matrixGPU, CoresAgentLin._matrixGPU, CoresLinTrans._matrixGPU, CoresIndiceNu._matrixGPU, _rho, _gamma, _gammahat);
	}
}


void PACGPU::updateQ()
{
	updateQAug << <_nAgent, _blockSize >> > (Q._matrixGPU, Qinit._matrixGPU, Xhat._matrixGPU, Muhat._matrixGPU, Nuhat._matrixGPU, nVoisin._matrixGPU, CoresAgentLin._matrixGPU, CoresLinTrans._matrixGPU, CoresIndiceNu._matrixGPU, _rhoInv, _rho, _gamma, augmente);
}

float PACGPU::updateRes(int indice)
{
	float resS = 0;
	float resR = 0;
	float resV = 0;
	
	float resTempS = X.max2(&Xhat);
	float resTempR = Muhat.max2(&Mu);
	float resTempV = Nuhat.max2(&Nu);

	if (resTempS > resS) {
		resS = resTempS;
	}
	if (resTempR > resR) {
		resR = resTempR;
	}
	if (resTempV > resV) {
		resV = resTempV;
	}
	//std::cout << "iter : " << indice << " " << resTempS << " " << resTempR << " " << resTempV << " | ";

	int numBlock = _nAgent;
	switch (_blockSize) {
	case 512:
		calcConstraintPAC<512> << <numBlock, _blockSize >> > (tempM._matrixGPU, tempM1._matrixGPU, X._matrixGPU, nVoisin._matrixGPU, CoresAgentLin._matrixGPU, CoresLinTrans._matrixGPU, CoresIndiceNu._matrixGPU);
		break;
	case 256:
		calcConstraintPAC<256> << <numBlock, _blockSize >> > (tempM._matrixGPU, tempM1._matrixGPU, X._matrixGPU, nVoisin._matrixGPU, CoresAgentLin._matrixGPU, CoresLinTrans._matrixGPU, CoresIndiceNu._matrixGPU);
		break;
	case 128:
		calcConstraintPAC<128> << <numBlock, _blockSize >> > (tempM._matrixGPU, tempM1._matrixGPU, X._matrixGPU, nVoisin._matrixGPU, CoresAgentLin._matrixGPU, CoresLinTrans._matrixGPU, CoresIndiceNu._matrixGPU);
		break;
	case 64:
		calcConstraintPAC< 64> << <numBlock, _blockSize >> > (tempM._matrixGPU, tempM1._matrixGPU, X._matrixGPU, nVoisin._matrixGPU, CoresAgentLin._matrixGPU, CoresLinTrans._matrixGPU, CoresIndiceNu._matrixGPU);
		break;
	case 32:
		calcConstraintPAC< 32> << <numBlock, _blockSize >> > (tempM._matrixGPU, tempM1._matrixGPU, X._matrixGPU, nVoisin._matrixGPU, CoresAgentLin._matrixGPU, CoresLinTrans._matrixGPU, CoresIndiceNu._matrixGPU);
		break;
	case 16:
		calcConstraintPAC< 16> << <numBlock, _blockSize >> > (tempM._matrixGPU, tempM1._matrixGPU, X._matrixGPU, nVoisin._matrixGPU, CoresAgentLin._matrixGPU, CoresLinTrans._matrixGPU, CoresIndiceNu._matrixGPU);
		break;
	case  8:
		calcConstraintPAC<  8> << <numBlock, _blockSize >> > (tempM._matrixGPU, tempM1._matrixGPU, X._matrixGPU, nVoisin._matrixGPU, CoresAgentLin._matrixGPU, CoresLinTrans._matrixGPU, CoresIndiceNu._matrixGPU);
		break;
	case  4:
		calcConstraintPAC<  4> << <numBlock, _blockSize >> > (tempM._matrixGPU, tempM1._matrixGPU, X._matrixGPU, nVoisin._matrixGPU, CoresAgentLin._matrixGPU, CoresLinTrans._matrixGPU, CoresIndiceNu._matrixGPU);
		break;
	case  2:
		calcConstraintPAC<  2> << <numBlock, _blockSize >> > (tempM._matrixGPU, tempM1._matrixGPU, X._matrixGPU, nVoisin._matrixGPU, CoresAgentLin._matrixGPU, CoresLinTrans._matrixGPU, CoresIndiceNu._matrixGPU);
		break;
	case  1:
		calcConstraintPAC<  1> << <numBlock, _blockSize >> > (tempM._matrixGPU, tempM1._matrixGPU, X._matrixGPU, nVoisin._matrixGPU, CoresAgentLin._matrixGPU, CoresLinTrans._matrixGPU, CoresIndiceNu._matrixGPU);
		break;
	}/**/
	resTempR = tempM1.max2();
	resTempV = tempM.max2();
	
	//std::cout << X.max2(&Xpre) << std::endl;

	resTempS = X.max2(&Xpre);
	
	//std::cout << " | " << resTempS << " " << resTempR << " " << resTempV << std::endl;

	
	if (resTempS > resS) {
		resS = resTempS;
	}
	if (resTempR > resR) {
		resR = resTempR;
	}
	if (resTempV > resV) {
		resV = resTempV;
	}

	
	resF.set(0, indice, resR);
	resF.set(1, indice, resS);
	resF.set(2, indice, resV);
	return MAX(MAX(resV, resS), resR);
}



void PACGPU::display() {

	std::cout << _name << std::endl;
}


__global__ void updateNuAug(float* Nu, float* Nuhat, float* Xhat, float* nVoisin, float* CoresAgentLin, float* CoresLinTrans, float* CoresindiceNu, float rho, float gamma, float theta) {

	int thIdx = threadIdx.x;
	int step = blockDim.x;
	int agent = blockIdx.x;
	int begin = CoresAgentLin[agent];
	int Mn = nVoisin[agent];
	int indiceNu = CoresindiceNu[begin];
	

	for (int lin = thIdx; lin < Mn; lin+= step) {

		float nuOld = Nu[indiceNu + lin];
		int linPeer = CoresLinTrans[begin + lin + 1];
		float nu = Nuhat[indiceNu + lin] + rho * gamma * (Xhat[begin + lin + 1 + Mn] - Xhat[linPeer]);
		
		
		Nu[indiceNu + lin] = nu;
		Nuhat[indiceNu + lin] = nu + theta * (nu - nuOld);

	}
}


__global__ void updateNuGPU(float* Nu, float* Nuhat, float* Xhat, float* nVoisin, float* CoresAgentLin, float* CoresLinTrans, float* CoresindiceNu, float rho, float gamma, float gammahat) {

	int thIdx = threadIdx.x;
	int step = blockDim.x;
	int agent = blockIdx.x;
	int begin = CoresAgentLin[agent];
	int Mn = nVoisin[agent];
	int indiceNu = CoresindiceNu[begin];

	for (int lin = thIdx; lin < Mn; lin += step) {

		float nuOld = Nu[indiceNu + lin];
		int linPeer = CoresLinTrans[begin + lin + 1];
		float dX = (Xhat[begin + lin + 1 + Mn] - Xhat[linPeer]);
		float nu = rho * gamma * dX + nuOld;
		Nu[indiceNu + lin] =  nu;
		Nuhat[indiceNu + lin] = nu + rho * gammahat * dX;
	}
}


__global__ void updateQAug(float* Q, float* Qinit, float* Xhat, float* Muhat, float* Nuhat, float* nVoisin, float* CoresAgentLin, float* CoresLinTrans, float* CoresIndiceNu, float rhoInv, float rho, float gamma, bool augmente) {
	int agent = blockIdx.x;
	int thIdx = threadIdx.x;
	int step = blockDim.x;

	int begin = CoresAgentLin[agent];
	int Mn = nVoisin[agent];
	int indiceNu = CoresIndiceNu[begin];
	int indiceMu = indiceNu + agent;
	float MuLocal = Muhat[indiceMu];
	
	if (thIdx == 0) {
		Q[begin] = Qinit[begin] + MuLocal - rhoInv * Xhat[begin];
	}

	for (int i = thIdx; i < Mn; i += step) {
		int lin = begin + i + 1;
		int indiceNuTrans = CoresIndiceNu[lin];
		float q  = Qinit[lin]      + Muhat[indiceMu + thIdx + 1] - MuLocal - Nuhat[indiceNuTrans] - rhoInv * Xhat[lin];
		float q2 = Qinit[lin + Mn] + Muhat[indiceMu + thIdx + 1] + Nuhat[indiceNu + i] - rhoInv * Xhat[lin + Mn];

		if (augmente) {	
			int linTrans = CoresLinTrans[lin];
			float tpi = Xhat[linTrans];
			q2 -= rho*gamma* tpi;
		}
		Q[lin] = q;
		Q[lin + Mn] = q2;

	}	
}



template <unsigned int blockSize>
__global__ void updateLocalProblPAC(float* X, float* Q, float* Hinv, float* matLb, float* matUb, float* CoresAgentLinBig, float* sizeQ, int sizeOPFmax) {

	// un bloc par ligne
	int thIdx = threadIdx.x;
	int step = blockDim.x;
	int l = blockIdx.x;
	float sum = 0;
	__shared__ float shArr[blockSize];
	int N = sizeQ[l];
	int indiceBegin = CoresAgentLinBig[l];
	for (int i = thIdx; i < N; i += step) {
		sum += Hinv[l * sizeOPFmax + i] * Q[indiceBegin + i];
	}

	shArr[thIdx] = sum;
	__syncthreads();

	if (blockSize >= 512) { if (thIdx < 256) { shArr[thIdx] += shArr[thIdx + 256]; } __syncthreads(); }
	if (blockSize >= 256) { if (thIdx < 128) { shArr[thIdx] += shArr[thIdx + 128]; } __syncthreads(); }
	if (blockSize >= 128) { if (thIdx < 64) { shArr[thIdx] += shArr[thIdx + 64]; } __syncthreads(); }
	if (thIdx < 32) {
		warpReduce<blockSize>(shArr, thIdx);
	}

	if (thIdx == 0) {
		float res = -shArr[0];
		
		float ub = matUb[l];
		float lb = matLb[l];
		X[l] = res + (ub - res) * (res > ub) + (lb - res) * (res < lb);
	}
}


__global__ void calculFcPAC(float* tempN1, float* tempNN, float* a, float* b, float* betaLin, float* X, float* CoresAgentLin, float* nVoisin) {
	int agent = blockIdx.x;
	int nAgent = gridDim.x;
	int thI = threadIdx.x;
	int step = blockDim.x;
	int begin = CoresAgentLin[agent];
	int Mn = nVoisin[agent];

	for (int i = thI; i < Mn; i += step) {
		tempNN[agent * nAgent + thI] = betaLin[begin + i + 1] * X[begin + i + 1];
	}
	
	if (thI == 0) {
		tempN1[agent] = X[begin] * (0.5*a[agent] * X[begin] + b[agent]);
	}

}






__global__ void updateP0PAC(float* matLb, float* matUb, float* Q, float* Qinit, float* Pmin, float* Pmax, float* Cost2, float* Lb, float* Ub, float* CoresAgentLin, float* nVoisin) {
	int agent = blockIdx.x;
	int thI = threadIdx.x;
	int step = blockDim.x;
	int begin = CoresAgentLin[agent];
	int Mn = nVoisin[agent];

	for (int lin = thI; lin < Mn; lin++) {
		int i = lin + begin;
		matLb[i] = Lb[agent];
		matLb[i + Mn] = -Ub[agent];
		matUb[i] = Ub[agent];
		matUb[i + Mn] = -Lb[agent];
	}

	if (thI == 0) {
		float q = Q[begin] - Qinit[begin];
		float a = Cost2[agent];
		Qinit[begin] = a;
		Q[begin] = q + a;
	}

}

