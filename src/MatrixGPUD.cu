#include "hip/hip_runtime.h"
#include "../head/MatrixGPUD.cuh" 

static const int warpSizeD = 32;



double MatrixGPUD::rand1()
{
    double a = (double)(rand()) / ((double)(RAND_MAX));
    return a;
}


///////////////////////////////////////////////////////////////////////////////
// Constructor
///////////////////////////////////////////////////////////////////////////////
MatrixGPUD::MatrixGPUD() {
#ifdef DEBUG_CONSTRUCTOR
    std::cout << "contructeur appele" << std::endl;
#endif
    _row = 0;
    _column = 0;
    _N = _row * _column;
    _numBlocks = ceil((_N + _blockSize - 1) / _blockSize);
}

MatrixGPUD::MatrixGPUD(int l, int c, double value, bool pos)
{
#ifdef DEBUG_CONSTRUCTOR
    std::cout << "contructeur parametre appele" << std::endl;
    std::cout << _matrixCPU << std::endl;
#endif
    _row = l;
    _column = c;
    _N = _row * _column;
    _numBlocks = ceil((_N + _blockSize - 1) / _blockSize);
    if (pos) {
        hipMalloc((void**)&_matrixGPU, sizeof(double) * _row * _column);
        setGPU << <_numBlocks, _blockSize >> > (_matrixGPU, value, _N);
        _GPU = true;
    }
    else {
        _matrixCPU = new double[l * c];
        for (int elem = 0; elem < l * c; elem++) {
            _matrixCPU[elem] = value;
        }
    }
#ifdef DEBUG_CONSTRUCTOR
    std::cout << _matrixGPU << std::endl;
#endif
}

MatrixGPUD::MatrixGPUD(const MatrixCPUD& m, bool pos)
{
    _row = m.getNLin();
    _column = m.getNCol();
    _N = _row * _column;
    _numBlocks = ceil((_N + _blockSize - 1) / _blockSize);

    if (pos) {
        _GPU = true;
        hipMalloc((void**)&_matrixGPU, sizeof(double) * _row * _column);
        hipMemcpy(_matrixGPU, m._matrixCPU, sizeof(double) * _row * _column, hipMemcpyHostToDevice);
    }
    else {
        _matrixCPU = new double[_row * _column];
        memcpy(_matrixCPU, m._matrixCPU, _row * _column * sizeof(double));
    }
    
}

MatrixGPUD::MatrixGPUD(const MatrixCPU& m, bool pos)
{
    _row = m.getNLin();
    _column = m.getNCol();
    _N = _row * _column;
    _numBlocks = ceil((_N + _blockSize - 1) / _blockSize);
    _matrixCPU = new double[_row * _column];
    for (int elem = 0; elem < _row * _column; elem++) {
        _matrixCPU[elem] = (double)m._matrixCPU[elem];
    }

    if (pos) {
        _GPU = true;
        hipMalloc((void**)&_matrixGPU, sizeof(double) * _row * _column);
        hipMemcpy(_matrixGPU, _matrixCPU, sizeof(double) * _row * _column, hipMemcpyHostToDevice);
    }
    
}

MatrixGPUD::MatrixGPUD(const MatrixGPUD & m)
{
#ifdef DEBUG_CONSTRUCTOR
    std::cout << "contructeur recopie appele" << std::endl;
#endif
    _row = m.getNLin();
    _column = m.getNCol();
    _N = _row * _column;
    _numBlocks = ceil((_N + _blockSize - 1) / _blockSize);

    if (m.getPos()) {
        hipMalloc((void**)&_matrixGPU, sizeof(double) * _row * _column);
        setGPU <<<_numBlocks, _blockSize >>> (_matrixGPU, m._matrixGPU, _N);
        _GPU = true;
    }
    else {
        _matrixCPU = new double[_row * _column];
        memcpy(_matrixCPU, m._matrixCPU, _row * _column * sizeof(double));
    }
}

MatrixGPUD& MatrixGPUD::operator=(const MatrixGPUD& m)
{
#ifdef DEBUG_CONSTRUCTOR
    std::cout << "contructeur operateur = appele" << std::endl;
#endif
    if (_row == m.getNLin() && _column == m.getNCol()) {
        //matrix already has the good size no free needed
        if (getPos()) {
            if (m.getPos()) {
                setGPU << <_numBlocks, _blockSize >> > (_matrixGPU, m._matrixGPU, _N);
            }
            else {
                hipMemcpy(_matrixGPU, m._matrixCPU, sizeof(double) * _row * _column, hipMemcpyHostToDevice);
            }
        }
        else {
            if (m.getPos()) {
                hipMemcpy(_matrixCPU, m._matrixGPU, sizeof(double) * _row * _column, hipMemcpyDeviceToHost);
            }
            else {
                memcpy(_matrixCPU, m._matrixCPU, _row * _column * sizeof(double));
            }
        }
    }
    else {
        _row = m.getNLin();
        _column = m.getNCol();
        _N = _row * _column;
        _numBlocks = ceil((_N + _blockSize - 1) / _blockSize);
        if (getPos()) {
            _GPU = false;
            hipFree(_matrixGPU);
        }
        else
        {
            DELETEA(_matrixCPU);
        }
        if (m.getPos()) {
            hipMalloc((void**)&_matrixGPU, sizeof(double) * _row * _column);
            setGPU << <_numBlocks, _blockSize >> > (_matrixGPU, m._matrixGPU, _N);
            _GPU = true;
        }
        else {
            _matrixCPU = new double[_row * _column];
            memcpy(_matrixCPU, m._matrixCPU, _row * _column * sizeof(double));
        }
    }
   
    return *this;
}

MatrixGPUD& MatrixGPUD::operator=(const MatrixCPUD& m)
{
    if (_row == m.getNLin() && _column == m.getNCol()) {
        //matrix already has the good size no free needed
        if (getPos()) {
            hipMemcpy(_matrixGPU, m._matrixCPU, sizeof(double) * _row * _column, hipMemcpyHostToDevice);
        }
        else {
            memcpy(_matrixCPU, m._matrixCPU, _row * _column * sizeof(double));
        }
    }
    else {
        _row = m.getNLin();
        _column = m.getNCol();
        _N = _row * _column;
        _numBlocks = ceil((_N + _blockSize - 1) / _blockSize);
        if (getPos()) {
            hipFree(_matrixGPU);
            hipMalloc((void**)&_matrixGPU, sizeof(double) * _row * _column);
            hipMemcpy(_matrixGPU, m._matrixCPU, sizeof(double) * _row * _column, hipMemcpyHostToDevice);
            _GPU = true;
        }
        else
        {
            DELETEA(_matrixCPU);
            _matrixCPU = new double[_row * _column];
            memcpy(_matrixCPU, m._matrixCPU, _row * _column * sizeof(double));
        } 
    }
    return *this;
}

void MatrixGPUD::preallocateReduction()
{
    if (preallocation) {
        hipHostFree(_preallocationFloat);
        hipFree(_preallocation);
        preallocation = false;
    }
    hipError_t c;
    do
    {
        c = hipHostAlloc(&_preallocationFloat, sizeof(double), hipHostMallocDefault);
     
        /*if (_preallocationFloat == nullptr) {
            
            std::cout << "prealocation echouer ? " << c << std::endl;
            std::cout <<  hipGetErrorName(c) << std::endl;
        }*/
    } while (c == 700);

    
    c = hipMalloc((void**)&_preallocation, sizeof(double) * _numBlocks);
    /*if (_preallocation == nullptr) {
        std::cout << _row << " " << _column << " " << _blockSize << std::endl;
        std::cout << "prealocation echouer ? " << c << " " << _numBlocks  <<std::endl;
        std::cout << hipGetErrorName(c) << std::endl;
    }*/
    if (c == 700) {
        exit(-1);
    }
    
    preallocation = true;
    setGPU <<<_numBlocks, _blockSize >>> (_preallocation, 0.0f, _numBlocks);
}

void MatrixGPUD::transferGPU()
{
    if (!_GPU) {
        if (!_matrixGPU) {
            hipMalloc((void**)&_matrixGPU, sizeof(double) * _row * _column);
        }
        hipMemcpy(_matrixGPU, _matrixCPU, sizeof(double) * _row * _column, hipMemcpyHostToDevice);
        //DELETEA(_matrixCPU);
        _GPU = true;
    }
    else {
        throw std::domain_error("already in the GPU");
    }
    
}

void MatrixGPUD::transferCPU()
{
    
    if (_GPU) {
        
        if (!_matrixCPU) {
            
            _matrixCPU = new double[_row * _column];
        }
        hipMemcpy(_matrixCPU, _matrixGPU, sizeof(double) * _row * _column, hipMemcpyDeviceToHost);
        //hipFree(_matrixGPU);
        //_matrixGPU = nullptr;
        _GPU = false;
    }
    else {
        std::cout << "already in the CPU" << _GPU <<std::endl;
        throw std::domain_error("already in the CPU");
    }

}

///////////////////////////////////////////////////////////////////////////////
// Getter
///////////////////////////////////////////////////////////////////////////////
 double MatrixGPUD::get(int i, int j, bool verbose) const
{
    //std::cout << "hey de taille " << _row << " " << _column << "pos "<< i <<" "<< j << std::endl;
    if ((i >= _row) || ( j >= _column) || (i < 0) || ( j < 0)) {
        throw std::out_of_range("index out of bounds");
    }
    if (_GPU) {
        double value;
        hipMemcpy(&value, _matrixGPU + i*_column+j, sizeof(double), hipMemcpyDeviceToHost);
        if (verbose) {
            std::cout << " Warning matrix on GPU" << std::endl;
        }
        return value;
        //throw std::invalid_argument("Matrix on GPU");
    }
    else {
        return _matrixCPU[i * _column + j];
    }
}

int MatrixGPUD::getNCol() const
{
    return _column;
}

int MatrixGPUD::getNLin() const
{
    return _row;
}

void MatrixGPUD::getCol(MatrixGPUD* col, int numCol, int offset)
{
    if (numCol < 0 || numCol >= _column) {
        throw std::out_of_range("index out of bounds");
    }
    if (offset < 0 || offset >= _row) {
        throw std::out_of_range("index out of bounds");
    }
    if (col->getNLin() != _row) {
        throw std::invalid_argument("not the same dimension");
    }
    if (col->getNCol() != 1) {
        throw std::invalid_argument("must be a column vector");
    }

    if (!_GPU && !col->getPos()) {
        for (int i = 0; i < offset; i++) {
            col->set(i, 0, 0);
        }
        for (int i = offset; i < _row; i++) {
            col->set(i, 0, get(i, numCol));
        }
    }
    else if (_GPU && col->getPos()) {
        setColGPU <<< _numBlocks, _blockSize >>> (col->_matrixGPU, _matrixGPU, numCol, _column, _row, offset);
    }

}

bool  MatrixGPUD::getPos() const
{
    return _GPU;
}
bool MatrixGPUD::dim(MatrixGPUD* m) const
{ 
    return ((_row == m->getNLin()) && (_column == m->getNCol()));
}


bool MatrixGPUD::isEqual(MatrixGPUD* m, double pre) const
{
    if (!dim(m)) {
        throw std::invalid_argument("not the same dimension");
    }
    else {
        if (_GPU || m->getPos()) {
            throw std::invalid_argument("Matrix on GPU");
        }
        else {
            for (int i = 0; i < _row; i++) {
                for (int j = 0; j < _column; j++) {
                    if (fabs(get(i, j) - m->get(i, j)) > pre) {
                        return false;
                    }
                }
            }
        }
    }
    return true;
}

void MatrixGPUD::toMatCPU(MatrixCPUD& m) const // passer m en paramètre
{
    if (m.getNCol() != _column || m.getNLin() != _row) {
        m.setSize(_row, _column);
    }
    if (_GPU) {
        hipMemcpy(m._matrixCPU, _matrixGPU, sizeof(double) * _row * _column, hipMemcpyDeviceToHost);
    }
    else {
        for (int i = 0; i < _row; i++) {
            for (int j = 0; j < _column; j++) 
            {
                m.set(i, j, get(i, j));
            }
        }
    }
}



void MatrixGPUD::setSize(int row, int column)
{
    _row = row;
    _column = column;
    _N = _row * _column;
    _numBlocks = ceil((_N + _blockSize - 1) / _blockSize);
    if (getPos()) {
        hipFree(_matrixGPU);
        hipMalloc((void**)&_matrixGPU, sizeof(double) * _row * _column);
        _GPU = true;
    }
    else
    {
        DELETEA(_matrixCPU);
        _matrixCPU = new double[_row * _column];
        memset(_matrixCPU, 0, _N * sizeof(double));
    }
}

///////////////////////////////////////////////////////////////////////////////
// Setter
///////////////////////////////////////////////////////////////////////////////
 void MatrixGPUD::set(int i, int j, double value, bool force)
{
    if ((i >= _row) || (j >= _column) || (i < 0) || (j < 0)) {
        throw std::out_of_range("index out of bounds");
    }
     if (_GPU && !force) {
        throw std::invalid_argument("Matrix on GPU");
     }
     else if (_GPU && force) {
         setGPUunique <<< 1, 1 >>> (_matrixGPU, value, i * _column + j);
     }
     else {
         //std::cout << "changement de valeur " << value << " en " << i << " " << j << std::endl;
         _matrixCPU[i * _column + j] = value;
     }
}

 void MatrixGPUD::setEyes(double value)
 {
     if (!_GPU) {
         int N = _row * (_row < _column) + _column * (_column <= _row);

         for (int i = 0; i < _row; i++) {
             for (int j = 0; j < _column; j++)
             {
                 if (i == j) {
                     set(i, j, value);
                 }
                 else {
                     set(i, j, 0);
                 }  
             }
         }
     }
     else {
         setEyesGPU<<< _numBlocks, _blockSize >>>(_matrixGPU, value, _column, _row);
     }
     
 }

 void MatrixGPUD::setEyes(MatrixGPUD* m)
 {
     if (m->getNLin() != _row || _row != _column || m->getNCol() != 1) 
     {
         throw std::invalid_argument("not the good dimension");
     }
     if (_GPU && m->getPos()) {
         setEyesGPU <<< _numBlocks, _blockSize >> > (_matrixGPU, m->_matrixGPU, _column, _row);

     }
     else if (!_GPU && !(m->getPos()))
     {
         for (int i = 0; i < _row; ++i)
         {
            this->set(i, i, m->get(i, 0));
         }
     }
     else {
         throw std::invalid_argument("Matrix not at the same place");
     }



 }

void MatrixGPUD::set(MatrixGPUD* m, bool synchrone, hipStream_t stream)
{
   
    if (!dim(m)) {
        throw std::invalid_argument("not the same dimension");
    }
    if (_GPU && m->getPos()) {
        if (synchrone) {
            setGPU <<<_numBlocks, _blockSize >> > (_matrixGPU, m->_matrixGPU, _N);
        }
        else {
            setGPU <<< _numBlocks, _blockSize, 0, stream>>> (_matrixGPU, m->_matrixGPU, _N);
        }
        
    }
    else if (!_GPU && !(m->getPos())) 
    {
        for (int i = 0; i < _row;++i)
        {
            for (int j = 0;j < _column;++j)
            { 
                this->set(i, j, m->get(i, j));
            }
        }
    }
    else {
        throw std::invalid_argument("Matrix not at the same place");
    }

}

void MatrixGPUD::set(MatrixCPUD* m)
{
    if (m->getNCol() != _column || m->getNLin() != _row) {
        throw std::invalid_argument("not the same dimension");
    }
    
    if (getPos()) {
        hipMemcpy(_matrixGPU, m->_matrixCPU, sizeof(double) * _row * _column, hipMemcpyHostToDevice);
    }
    else {
        memcpy(_matrixCPU, m->_matrixCPU, _row * _column * sizeof(double));
    }
   
    
}

void MatrixGPUD::setTrans(MatrixGPUD* m)
{
    if (_column != m->getNLin() || _row != m->getNCol()) {
        throw std::invalid_argument("not the same transposed dimension");
    }
    if (getPos() && m->getPos()) {
        setTransGPU << <_numBlocks, _blockSize >> > (_matrixGPU, m->_matrixGPU, _column, _row);
    }
    else if (!getPos() && !(m->getPos()))
    {
        for (int i = 0; i < _row; i++) {
            for (int j = 0; j < _column; j++) {
                set(i, j, m->get(j, i));
            }
        }
    }
    else {
        throw std::invalid_argument("Matrix not at the same place");
    }

}


void MatrixGPUD::setRand(double eps)
{
    //exit(1);
    if (_GPU) {

        /*hiprandGenerator_t gen;
        hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
        hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL);
        hiprandGenerateUniform(gen, _matrixGPU, _N);
        hiprandDestroyGenerator(gen);*/
        
        hiprandState* state = nullptr;
        hipMalloc((void**)&state, _N * sizeof(hiprandState));
        setup_kernelD <<<_numBlocks, _blockSize >>> (state);
        generate_kernel << <_numBlocks, _blockSize >> > (state, _matrixGPU, eps, _N);
        //throw std::invalid_argument("Matrix on GPU");
    }
    else {
        int N = _column * _row;
        for (int elem = 0; elem < N; elem++) {
            _matrixCPU[elem] = 2 * (rand1() - 0.5) * eps;

        }
    }
    
}

void MatrixGPUD::setBloc(int iBegin, int iEnd, int jBegin, int jEnd, MatrixGPUD* m)
{
    if ((iBegin < 0) || (jBegin < 0) || iEnd > _row || jEnd > _column) {
        throw std::out_of_range("index out of bounds");
    } if ((iBegin > iEnd) || (jBegin > jEnd)) {
        throw std::invalid_argument("xBegin must be smaller than xEnd");
    } if (m->getNLin() != (iEnd - iBegin) || m->getNCol() != (jEnd - jBegin)) {
        throw std::invalid_argument("not the same dimension");
    }
    if (!_GPU && !(m->getPos())) {
        int row = 0;

        for (int i = iBegin; i < iEnd; i++) {
            int col = 0;
            for (int j = jBegin; j < jEnd;j++) {
                set(i, j, m->get(row, col));
                col = col + 1;
            }
            row = row + 1;
        }
    }
    else if (getPos() && (m->getPos())) {
        /*const int nThread = 16;
         const int bx = (jEnd - jBegin + nThread - 1) / nThread;
         const int by = (iEnd - iBegin + nThread - 1) / nThread;
         dim3 gridBlock(bx, by);
         dim3 dimBlock(nThread, nThread);*/
        SetBlocGPU << <m->_numBlocks, m->_blockSize >> > (_matrixGPU, m->_matrixGPU, iBegin, iEnd, jBegin, jEnd, _column);
    }
    else {
       
        throw std::invalid_argument("Matrix not at the same place");
    }
}



void MatrixGPUD::setBloc(int iBegin, int iEnd, int jBegin, int jEnd, MatrixGPUD* m, double factor)
{
    if ((iBegin < 0) || (jBegin < 0) || iEnd > _row || jEnd > _column) {
        throw std::out_of_range("index out of bounds");
    } if ((iBegin > iEnd) || (jBegin > jEnd)) {
        throw std::invalid_argument("xBegin must be smaller than xEnd");
    } if (m->getNLin() != (iEnd - iBegin) || m->getNCol() != (jEnd - jBegin)) {
        throw std::invalid_argument("not the same dimension");
    }
    if (!_GPU && !(m->getPos())) {
        int row = 0;

        for (int i = iBegin; i < iEnd; i++) {
            int col = 0;
            for (int j = jBegin; j < jEnd; j++) {
                set(i, j, factor * m->get(row, col));
                col = col + 1;
            }
            row = row + 1;
        }
    }
    else if (getPos() && (m->getPos())) {
        /*const int nThread = 16;
        const int bx = (jEnd - jBegin + nThread - 1) / nThread;
        const int by = (iEnd - iBegin + nThread - 1) / nThread;
        dim3 gridBlock(bx, by);
        dim3 dimBlock(nThread, nThread);*/
        SetBlocGPU <<<m->_numBlocks, m->_blockSize >> > (_matrixGPU, m->_matrixGPU, iBegin, iEnd, jBegin, jEnd, _column, factor);
    }
    else {
        throw std::invalid_argument("Matrix not at the same place");
    }
}
void MatrixGPUD::setBloc(int iBegin, int iEnd, int jBegin, int jEnd, MatrixCPUD* m)
{
    if ((iBegin < 0) || (jBegin < 0) || iEnd > _row || jEnd > _column) {
        throw std::out_of_range("index out of bounds");
    } if ((iBegin > iEnd) || (jBegin > jEnd)) {
        throw std::invalid_argument("xBegin must be smaller than xEnd");
    } if (m->getNLin() != (iEnd - iBegin) || m->getNCol() != (jEnd - jBegin)) {
        throw std::invalid_argument("not the same dimension");
    }
    if (!_GPU) {
        int row = 0;

        for (int i = iBegin; i < iEnd; i++) {
            int col = 0;
            for (int j = jBegin; j < jEnd;j++) {
                set(i, j, m->get(row, col));
                col = col + 1;
            }
            row = row + 1;
        }
    }
    else {
        throw std::domain_error("Matrix on GPU");
    }
}


void MatrixGPUD::swap(MatrixGPUD* m)
{
    if (!dim(m)) {
        throw std::invalid_argument("not the same dimension");
    }
    if (_GPU && m->getPos()) {
        double* temp = _matrixGPU;
        _matrixGPU = m->_matrixGPU;
        m->_matrixGPU = temp;

    }
    else if (!_GPU && !(m->getPos())) {
        double* temp = _matrixCPU;
        _matrixCPU = m->_matrixCPU;
        m->_matrixCPU = temp;
    }
    else {
        throw std::invalid_argument("Matrix not at the same place");
    }

    
    
}

void MatrixGPUD::replace(double previous, double newValue)
{
    if (_GPU) {
        replaceGPU <<<_numBlocks, _blockSize >> > (_matrixGPU, previous, newValue, _N);
    }
    else {
        for (int i = 0;i < _row;++i)
        {
            for (int j = 0;j < _column;++j)
            {
                if (get(i, j) == previous) {
                    this->set(i, j, newValue);
                }
            }
        }
    }
}

///////////////////////////////////////////////////////////////////////////////
// Addition
///////////////////////////////////////////////////////////////////////////////
void MatrixGPUD::add(MatrixGPUD* m)
{
    if (!dim(m)) {
        throw std::invalid_argument("not the same dimension");
    }
    if (_GPU && m->getPos()) 
    {
        addGPU<<<_numBlocks, _blockSize>>>(_matrixGPU,m->_matrixGPU,_N);
    }
    else if (!_GPU && !(m->getPos()))
    {
        for (int i = 0;i < _row;++i)
        {
            for (int j = 0;j < _column;++j)
            {
                double r = get(i, j) + m->get(i, j);
                this->set(i, j, r);
            }
        }
    } else {
        throw std::invalid_argument("Matrix not at the same place");
    } 
}

void MatrixGPUD::addVector(MatrixGPUD* v)
{
    if (((v->getNCol() != 1) || (v->getNLin() != _row)) && ((v->getNLin() != 1) || (v->getNCol() != _column))) {
        throw std::invalid_argument("wrong dimension of the vector");
    }
    if (v->getNCol() == 1) {
        if (_GPU && v->getPos()) 
        {
            addVectorGPU1<<<_numBlocks, _blockSize >>>(_matrixGPU, v->_matrixGPU, _column, _N);
        }
        else if ((!_GPU) && !(v->getPos())) {
            for (int i = 0;i < _row;++i)
            {
                for (int j = 0;j < _column;++j)
                {
                    double r = get(i, j) + v->get(i, 0);
                    this->set(i, j, r);
                }
            }
        }
        else {
            throw std::invalid_argument("Matrix not at the same place");
        } 
    }
    else {
        if (_GPU && v->getPos())
        {
            addVectorGPU2<<<_numBlocks, _blockSize >>>(_matrixGPU, v->_matrixGPU, _column, _N);
        }
        else if ((!_GPU) && !(v->getPos())) {
            for (int i = 0;i < _row;++i)
            {
                for (int j = 0;j < _column;++j)
                {
                    double r = get(i, j) + v->get(0, j);
                    this->set(i, j, r);
                }
            }
        }
        else {
            throw std::invalid_argument("Matrix not at the same place");
        }
    }
}
void MatrixGPUD::add(double c)
{
    if (_GPU) {
        addGPU<<<_numBlocks,_blockSize >>>(_matrixGPU,c, _N);
    }
    else {
        for (int i = 0;i < _row;++i)
        {
            for (int j = 0;j < _column;++j)
            {
                double r = get(i, j) + c;
                this->set(i, j, r);
            }
        }
    }
}

void MatrixGPUD::add(MatrixGPUD* m1, MatrixGPUD* m2)
{
    if (!m1->dim(m2)) {
        throw std::invalid_argument("not the same dimension");
    }
    if (!dim(m1)) {
        throw std::invalid_argument("not the same dimension");
    }
    if (_GPU && m1->getPos() && m2->getPos()) 
    {
        addGPU<<<_numBlocks, _blockSize >> > (_matrixGPU, m1->_matrixGPU,m2->_matrixGPU, _N);
    }
    else if (!_GPU && !(m1->getPos()) && !(m2->getPos()))
    {
        for (int i = 0;i < _row;++i)
        {
            for (int j = 0;j < _column;++j)
            {
                double r = m1->get(i, j) + m2->get(i, j);
                this->set(i, j, r);
            }
        }
    }
    else {
        throw std::invalid_argument("Matrix not at the same place");
    }
    
}
void MatrixGPUD::add(MatrixGPUD* m, double c)
{
    if (_GPU && m->getPos()) 
    {
        addGPU<<<_numBlocks, _blockSize >>> (_matrixGPU,m->_matrixGPU, c, _N);
    }
    else if ((!_GPU) && !(m->getPos())) 
    {
        for (int i = 0;i < _row;++i)
        {
            for (int j = 0;j < _column;++j)
            {
                double r = m->get(i, j) + c;
                this->set(i, j, r);
            }
        }
    }
    else {
        throw std::invalid_argument("Matrix not at the same place");
    }
    

}
void MatrixGPUD::addTrans(MatrixGPUD* m)
{
    MatrixGPUD temp(*this);
    if (_row != m->getNCol() && _column != m->getNLin())
    {
        throw std::invalid_argument("not the same dimension (transpose)");
    }
    if (_GPU && m->getPos())
    {
        addTransGPU<<<_numBlocks, _blockSize >>>(temp._matrixGPU, _matrixGPU, m->_matrixGPU,_column,_row,_N);
    }
    else if (!_GPU && !(m->getPos()))
    {
        for (int i = 0;i < _row;++i)
        {
            for (int j = 0;j < _column;++j)
            {
                double r = get(i, j) + m->get(j, i);
                temp.set(i, j, r);
            }
        }
    }
    else {
        throw std::invalid_argument("Matrix not at the same place");
    }
    this->set(&temp);
    
}
///////////////////////////////////////////////////////////////////////////////
// subtraction
///////////////////////////////////////////////////////////////////////////////
void MatrixGPUD::subtract(MatrixGPUD* m1, MatrixGPUD* m2)
{
    if (!m1->dim(m2)) {
        throw std::invalid_argument("not the same dimension");
        
    }
    if (!dim(m1)) {
        throw std::invalid_argument("not the same dimension");
    }
    if (_GPU && m1->getPos() && m2->getPos())
    {
        substractGPU<<<_numBlocks, _blockSize >> > (_matrixGPU, m1->_matrixGPU, m2->_matrixGPU, _N);
    }
    else if (!_GPU && !(m1->getPos()) && !(m2->getPos()))
    {
        for (int i = 0;i < _row;++i)
        {
            for (int j = 0;j < _column;++j)
            {
                double r = m1->get(i, j) - m2->get(i, j);
                this->set(i, j, r);
            }
        }
    }
    else {
        throw std::invalid_argument("Matrix not at the same place");
    }
    
}
void MatrixGPUD::subtract(MatrixGPUD* m)
{
    
    if (!dim(m)) {
        throw std::invalid_argument("not the same dimension");
    }
    if (_GPU && m->getPos())
    {
        substractGPU <<<_numBlocks, _blockSize >> > ( _matrixGPU, m->_matrixGPU, _N);
    }
    else if (!_GPU && !(m->getPos()))
    {
        for (int i = 0;i < _row;++i)
        {
            for (int j = 0;j < _column;++j)
            {
                double r = get(i, j) - m->get(i, j);
                set(i, j, r);
            }
        }
    }
    else {
        throw std::invalid_argument("Matrix not at the same place");
    }
   
    
}
void MatrixGPUD::subtractVector(MatrixGPUD* v)
{
    if (((v->getNCol() != 1) || (v->getNLin() != _row)) && ((v->getNLin() != 1) || (v->getNCol() != _column))) {
        throw std::invalid_argument("wrong dimension of the vector");
    }
    if (v->getNCol() == 1) {
        if (_GPU && v->getPos())
        {
            substractVectorGPU1 <<<_numBlocks, _blockSize >>>(_matrixGPU, v->_matrixGPU, _column, _N);
        }
        else if ((!_GPU) && !(v->getPos())) {
            for (int i = 0;i < _row;++i)
            {
                for (int j = 0;j < _column;++j)
                {
                    double r = get(i, j) - v->get(i, 0);
                    this->set(i, j, r);
                }
            }
        }
        else {
            throw std::invalid_argument("Matrix not at the same place");
        }
    }
    else {
        if (_GPU && v->getPos())
        {
            substractVectorGPU2 <<<_numBlocks, _blockSize >>>(_matrixGPU, v->_matrixGPU, _column, _N);
        }
        else if ((!_GPU) && !(v->getPos())) {
            for (int i = 0;i < _row;++i)
            {
                for (int j = 0;j < _column;++j)
                {
                    double r = get(i, j) - v->get(0, j);
                    this->set(i, j, r);
                }
            }
        }
        else {
            throw std::invalid_argument("Matrix not at the same place");
        }
    }

}
void MatrixGPUD::subtractTrans(MatrixGPUD* m)
{
    if (_row != m->getNCol() && _column != m->getNLin())
    {
        throw std::invalid_argument("not the same dimension (transpose)");
    }
    MatrixGPUD temp(*this);
    if (_GPU && m->getPos())
    {
        substractTransGPU <<<_numBlocks, _blockSize >>>(temp._matrixGPU, _matrixGPU, m->_matrixGPU, _column, _row, _N);
    }
    else if (!_GPU && !(m->getPos()))
    {
        for (int i = 0;i < _row;++i)
        {
            for (int j = 0;j < _column;++j)
            {
                double r = get(i, j) - m->get(j, i);
                temp.set(i, j, r);
            }
        }
    }
    else {
        throw std::invalid_argument("Matrix not at the same place");
    }
    this->set(&temp);
}

///////////////////////////////////////////////////////////////////////////////
// Multiplication
///////////////////////////////////////////////////////////////////////////////


void MatrixGPUD::multiply(double c)
{
    if (_GPU) {
        multiplyGPU<<<_numBlocks, _blockSize >>> (_matrixGPU, c, _N);
    }
    else {
        for (int i = 0;i < _row;++i)
        {
            for (int j = 0;j < _column;++j)
            {
                double r = get(i, j) * c;
                this->set(i, j, r);
            }
        }
    }
        
}

void MatrixGPUD::multiplyMat(MatrixGPUD* A, MatrixGPUD* B)
{
    if (A->getNLin() != getNLin()) {
        throw std::invalid_argument("result must be compatible with A (row)");
    }
    else if (A->getNCol() != B->getNLin()) {
        throw std::invalid_argument("A must be compatible with B (column with row)");
    }
    else if (getNCol() != B->getNCol()) {
        throw std::invalid_argument("result must be compatible with B (column)");

    }
    if (_GPU && A->getPos() && B->getPos()) { // solution temporaire
        transferCPU();
        A->transferCPU();
        B->transferCPU();
        double r = 0;
        int p = A->getNCol();
        for (int i = 0; i < _row; ++i)
        {
            for (int j = 0; j < _column; ++j)
            {
                r = 0;
                for (int k = 0; k < p; ++k)
                {
                    r += A->get(i, k) * B->get(k, j);
                }
                this->set(i, j, r);
            }
        }
        transferGPU();
        A->transferGPU();
        B->transferGPU();
    }
    else if (!_GPU && !(A->getPos()) && !B->getPos()) {
        double r = 0;
        int p = A->getNCol();
        for (int i = 0; i < _row; ++i)
        {
            for (int j = 0; j < _column; ++j)
            {
                r = 0;
                for (int k = 0; k < p; ++k)
                {
                    r += A->get(i, k) * B->get(k, j);
                }
                this->set(i, j, r);
            }
        }
    }
    else {
        throw std::invalid_argument("Matrix not at the same place");
    }
}

void MatrixGPUD::multiply(MatrixGPUD* Mat, MatrixGPUD* vect, bool trans)
{
    // result = Mat*vect  nLine*nCol = nLine*Taille *Taille * Ncol 
    if (trans) {
        if (getNLin() != 1) {
            throw std::invalid_argument("result must be a row vector ");
        }
        else if (getNCol() != Mat->getNLin()) {
            throw std::invalid_argument("result must be compatible with Mat");
        }
        else if (vect->getNLin() != 1) {
            throw std::invalid_argument("vect must be a row vector ");
        }
        else if (vect->getNCol() != Mat->getNCol()) {
            throw std::invalid_argument("vect must be compatible with Mat");
        }
    }
    else {
        if (getNCol() != 1) {
            throw std::invalid_argument("result must be a column vector ");
        }
        else if (getNLin() != Mat->getNLin()) {
            throw std::invalid_argument("result must have the same row number as the Mat");
        }
        else if (vect->getNCol() != 1) {
            throw std::invalid_argument("vect must be a column vector ");
        }
        else if (vect->getNLin() != Mat->getNCol()) {
            throw std::invalid_argument("vect must be compatible with Mat");
        }
    }
   
    if (_GPU && Mat->getPos() && vect->getPos())
    {
        int numBlock = Mat->getNLin();
        switch (_blockSize) {
        case 512:
            multiplyGPU<512> << <numBlock, _blockSize >> > (_matrixGPU, Mat->_matrixGPU, vect->_matrixGPU, Mat->getNCol());
            break;
        case 256:
            multiplyGPU<256> << <numBlock, _blockSize >> > (_matrixGPU, Mat->_matrixGPU, vect->_matrixGPU, Mat->getNCol());
            break;
        case 128:
            multiplyGPU<128> << <numBlock, _blockSize >> > (_matrixGPU, Mat->_matrixGPU, vect->_matrixGPU, Mat->getNCol());
            break;
        case 64:
            multiplyGPU< 64> << <numBlock, _blockSize >> > (_matrixGPU, Mat->_matrixGPU, vect->_matrixGPU, Mat->getNCol());
            break;
        case 32:
            multiplyGPU< 32> << <numBlock, _blockSize >> > (_matrixGPU, Mat->_matrixGPU, vect->_matrixGPU, Mat->getNCol());
            break;
        case 16:
            multiplyGPU< 16> << <numBlock, _blockSize >> > (_matrixGPU, Mat->_matrixGPU, vect->_matrixGPU, Mat->getNCol());
            break;
        case  8:
            multiplyGPU<  8> << <numBlock, _blockSize >> > (_matrixGPU, Mat->_matrixGPU, vect->_matrixGPU, Mat->getNCol());
            break;
        case  4:
            multiplyGPU<  4> << <numBlock, _blockSize >> > (_matrixGPU, Mat->_matrixGPU, vect->_matrixGPU, Mat->getNCol());
            break;
        case  2:
            multiplyGPU<  2> << <numBlock, _blockSize >> > (_matrixGPU, Mat->_matrixGPU, vect->_matrixGPU, Mat->getNCol());
            break;
        case  1:
            multiplyGPU<  1> << <numBlock, _blockSize >> > (_matrixGPU, Mat->_matrixGPU, vect->_matrixGPU, Mat->getNCol());
            break;
        }
        
    }
    else if (!_GPU && !(Mat->getPos()) && !vect->getPos())
    {
        if (trans) {
            for (int i = 0; i < Mat->getNLin(); ++i)
            {
                double sum = 0;
                for (int j = 0; j < Mat->getNCol(); ++j)
                {
                    sum += Mat->get(i, j) * vect->get(0, j);
                }
                set(0, i, sum);
            }
        }
        else {
            for (int i = 0; i < _row; ++i)
            {
                double sum = 0;
                for (int j = 0; j < Mat->getNCol(); ++j)
                {
                    sum += Mat->get(i, j) * vect->get(j, 0);
                }
                set(i, 0, sum);
            }
        }
    }
    else {
        throw std::invalid_argument("Matrix not at the same place");
    }
}

void MatrixGPUD::linearOperation(MatrixGPUD* A, MatrixGPUD* x, MatrixGPUD* b, bool trans)
{
    if (trans) {
        if (getNLin() != 1) {
            throw std::invalid_argument("result must be a row vector ");
        }
        else if (getNCol() != A->getNLin()) {
            throw std::invalid_argument("result must be compatible with A");
        }
        else if (x->getNLin() != 1 || b->getNLin() != 1) {
            throw std::invalid_argument("x and b must be a row vector ");
        }
        else if (x->getNCol() != A->getNCol()) {
            throw std::invalid_argument("x must be compatible with A");
        }
    }
    else {
        if (getNCol() != 1) {
            throw std::invalid_argument("result must be a column vector ");
        }
        else if (getNLin() != A->getNLin()) {
            throw std::invalid_argument("result must have the same row number as A");
        }
        else if (x->getNCol() != 1 || b->getNCol() != 1) {
            throw std::invalid_argument("x and b must be a column vector ");
        }
        else if (x->getNLin() != A->getNCol()) {
            throw std::invalid_argument("x must be compatible with Mat");
        }
    }

    if (_GPU && A->getPos() && b->getPos() && x->getPos())
    {
        int numBlock = A->getNLin();
        switch (_blockSize) {
        case 512:
            linearOpGPU<512> << <numBlock, _blockSize >> > (_matrixGPU, A->_matrixGPU, x->_matrixGPU, b->_matrixGPU, A->getNCol());
            break;
        case 256:
            linearOpGPU<256> << <numBlock, _blockSize >> > (_matrixGPU, A->_matrixGPU, x->_matrixGPU, b->_matrixGPU, A->getNCol());
            break;
        case 128:
            linearOpGPU<128> << <numBlock, _blockSize >> > (_matrixGPU, A->_matrixGPU, x->_matrixGPU, b->_matrixGPU, A->getNCol());
            break;
        case 64:
            linearOpGPU< 64> << <numBlock, _blockSize >> > (_matrixGPU, A->_matrixGPU, x->_matrixGPU, b->_matrixGPU, A->getNCol());
            break;
        case 32:
            linearOpGPU< 32> << <numBlock, _blockSize >> > (_matrixGPU, A->_matrixGPU, x->_matrixGPU, b->_matrixGPU, A->getNCol());
            break;
        case 16:
            linearOpGPU< 16> << <numBlock, _blockSize >> > (_matrixGPU, A->_matrixGPU, x->_matrixGPU, b->_matrixGPU, A->getNCol());
            break;
        case  8:
            linearOpGPU<  8> << <numBlock, _blockSize >> > (_matrixGPU, A->_matrixGPU, x->_matrixGPU, b->_matrixGPU, A->getNCol());
            break;
        case  4:
            linearOpGPU<  4> << <numBlock, _blockSize >> > (_matrixGPU, A->_matrixGPU, x->_matrixGPU, b->_matrixGPU, A->getNCol());
            break;
        case  2:
            linearOpGPU<  2> << <numBlock, _blockSize >> > (_matrixGPU, A->_matrixGPU, x->_matrixGPU, b->_matrixGPU, A->getNCol());
            break;
        case  1:
            linearOpGPU<  1> << <numBlock, _blockSize >> > (_matrixGPU, A->_matrixGPU, x->_matrixGPU, b->_matrixGPU, A->getNCol());
            break;
        }

    }
    else if (!_GPU && !(A->getPos()) && !x->getPos() && !b->getPos())
    {
        if (trans) {
            for (int i = 0; i < A->getNLin(); ++i)
            {
                double sum = 0;
                for (int j = 0; j < A->getNCol(); ++j)
                {
                    sum += A->get(i, j) * x->get(0, j);
                }
                set(0, i, sum + b->get(0,i));
            }
        }
        else {
            for (int i = 0; i < _row; ++i)
            {
                double sum = 0;
                for (int j = 0; j < A->getNCol(); ++j)
                {
                    sum += A->get(i, j) * x->get(j, 0);
                }
                set(i, 0, sum + b->get(i,0));
            }
        }
    }
    else {
        throw std::invalid_argument("Matrix not at the same place");
    }
}

///////////////////////////////////////////////////////////////////////////////
// Multiplication Terme � Terme
///////////////////////////////////////////////////////////////////////////////

void MatrixGPUD::multiplyT(MatrixGPUD* m)
{
    if (!dim(m)) {
        throw std::invalid_argument("not the same dimension");
    }
    if (_GPU && m->getPos())
    {
        multiplyTGPU <<<_numBlocks, _blockSize >>> (_matrixGPU, m->_matrixGPU, _N);
    }
    else if (!_GPU && !(m->getPos()))
    {
        for (int i = 0;i < _row;++i)
        {
            for (int j = 0;j < _column;++j)
            {
                double r = get(i, j) * m->get(i, j);
                this->set(i, j, r);
            }
        }
    }
    else {
        throw std::invalid_argument("Matrix not at the same place");
    }
}

void MatrixGPUD::multiplyT(MatrixGPUD* m1, MatrixGPUD* m2)
{
    if (!m1->dim(m2)) {
        throw std::invalid_argument("not the same dimension");
    }
    if (!dim(m1)) {
        throw std::invalid_argument("not the same dimension");
    }
    if (_GPU && m1->getPos() && m2->getPos())
    {
        multiplyTGPU<<<_numBlocks, _blockSize >>>(_matrixGPU, m1->_matrixGPU, m2->_matrixGPU, _N);
    }
    else if (!_GPU && !(m1->getPos()) && !(m2->getPos()))
    {
        for (int i = 0;i < _row;++i)
        {
            for (int j = 0;j < _column;++j)
            {
                double r = m1->get(i, j) * m2->get(i, j);
                this->set(i, j, r);
            }
        }
    }
    else {
        throw std::invalid_argument("Matrix not at the same place");
    }
}


void MatrixGPUD::divide(double c)
{
    if (c == 0) {
        throw std::domain_error("divide by 0");
    }
    if (_GPU) {
        divideGPU <<<_numBlocks, _blockSize >>> (_matrixGPU, c, _N);
    }
    else {
        for (int i = 0;i < _row;++i)
        {
            for (int j = 0;j < _column;++j)
            {
                double r = get(i, j) / c;
                this->set(i, j, r);
            }
        }
    }
    
}

void MatrixGPUD::divideT(MatrixGPUD* m)
{
    
    if (!dim(m)) {
        throw std::invalid_argument("not the same dimension");
    }
    if (_GPU && m->getPos())
    {
        divideGPU<<<_numBlocks, _blockSize >>>(_matrixGPU, m->_matrixGPU, _N);
    }
    else if (!_GPU && !(m->getPos()))
    {
        MatrixGPUD temp(*this);
        double r = 0;
        double f = 0;
        for (int i = 0;i < _row;++i)
        {
            for (int j = 0;j < _column;++j)
            {
                f = m->get(i, j);
                if (f == 0) {
                    throw std::domain_error("divide by 0");
                }
                r = get(i, j) / f;
                temp.set(i, j, r);
            }
        }
        set(&temp);
    }
    else {
        throw std::invalid_argument("Matrix not at the same place");
    }
    
}

void MatrixGPUD::invertGaussJordan(MatrixGPUD* mToInvert)
{
    
    if (!dim(mToInvert)) {
        throw std::invalid_argument("not the same dimension");
    }
    if (_row != _column) {
        throw std::invalid_argument("must be a square matrix");
    }
   
    if (!_GPU && !mToInvert->getPos()) {
        MatrixCPUD m;
        mToInvert->toMatCPU(m);
       
        MatrixCPUD augmented(_row, _column);
        augmented.setEyes(1);
        MatrixCPUD indices(1, 2);
        int r = 0;
        for (int column = 0; column < _column; column++) {
            
            double pivotabs = m.maxAbs(r, _row, column, column + 1, &indices);
            int k = indices.get(0, 0); // indice max de la colonne j
            double pivot = m.get(k, column);
            if (pivotabs < 0.000001f) {
                throw std::invalid_argument("not invertible matrix");
            }
            else {
                
                for (int j = 0; j < _column; j++) {
                    augmented.set(k, j, augmented.get(k, j) / pivot);
                    m.set(k, j, m.get(k, j) / pivot);
                }
                
                if (k != r) {
                    augmented.swapLine(k, r);
                    m.swapLine(k, r);
                }
                
                for (int i = 0; i < _row; i++) {
                    if (i != r) {
                        double local = m.get(i, column);
                        m.subtractRow(i, r, local);
                        augmented.subtractRow(i, r, local);
                    }
                }
                r++;
            }
        }
        
        set(&augmented);
       
    }
   
    else if (_GPU && mToInvert->getPos()) {
        MatrixGPUD m(*mToInvert);
        setEyes(1);
        dim3 threadsPerBlock(32, 32);
        int r = 0;
        MatrixGPUD matCol(_row, 1, 0, 1);
        int k = 0;
        
        for (int column = 0; column < _column; column++) {
           
            //transferCPU();
            ///display();
            //transferGPU();

            m.getCol(&matCol, column, r);
            double pivotAbs = matCol.max2(&k); // comme matCol est un vecteur colonne, la position du maximum correspond directement à la ligne !
            double pivot = matCol.get(k, 0, false);
            
            
            if (pivotAbs < 0.000001f) {
                std::cout << "not invertible " << column <<" " << pivotAbs << std::endl;
                if (_N < 100) {
                    mToInvert->display(true);
                }
                throw std::invalid_argument("not invertible matrix");
            }
            else {
                normalisationGJ << <_numBlocks, _blockSize >> > (_matrixGPU, k, _column, pivot); // normalisation pour toute la ligne k
                normalisationGJ <<<_numBlocks, _blockSize >>> (m._matrixGPU, k, _column, pivot);
                
              
                if (k != r) {
                    swapLineGJ << <_numBlocks, _blockSize >> > (_matrixGPU, k, r, _column);// swap des lignes k et r 
                    swapLineGJ << <_numBlocks, _blockSize >> > (m._matrixGPU, k, r, _column);// swap des lignes k et r 
                }
                
                // soustration de la ligne sauf pour la r
                eliminationGJ <<<_row, _blockSize >> > (m._matrixGPU, _matrixGPU, r, _row, _column);
                r++;
            }
        }
    }
    else {
        throw std::invalid_argument("Matrix not at the same place");
    }


}

void MatrixGPUD::LUPFactorization(MatrixGPUD* A, MatrixGPUD* P)
{
    double Tol = 0.0000001;
    int n = getNLin();
    A->set(this);

    // code from wikipedia adapted
    if (getNCol() != getNLin()) {
        throw std::invalid_argument("A must be square");
    }
    if (P->getNCol() != 1 || P->getNLin() != (getNCol() + 1)) {
        throw std::invalid_argument("wrong size of P");
    }

    if (!_GPU && !A->getPos() && !P->getPos()) {
        
        for (int i = 0; i < n; i++) {
            P->set(i, 0, i); //Unit permutation matrix, P[N] initialized with N
        }

        double absA = 0;
        int j = 0;
        for (int col = 0; col < n; col++) {
            double maxA = 0.0;
            int imax = col;
            for (int k = col; k < n; k++) {
                absA = fabs(get(k, col));
                if (absA > maxA)
                {
                    maxA = absA;
                    imax = k;
                }
            }
            if (maxA < Tol) {
                throw std::invalid_argument("matrix is degenerate");
            }

            if (imax != col) { //le max pas sur la diagonal
                //pivoting P
                j = P->get(col, 0);
                P->set(col, 0, P->get(imax, 0));
                P->set(imax, 0, j);

                //pivoting rows of A
                A->swapLine(col, imax);

                //counting pivots starting from N (for determinant)
                P->set(n, 0, P->get(n, 0) + 1);
            }


            for (int i = col + 1; i < n; i++) {

                A->set(i, col, A->get(i, col) / A->get(col, col)); //A[j][i] /= A[i][i];


                for (int k = col + 1; k < n; k++) {
                    A->set(i, k, A->get(i, k) - A->get(i, col) * A->get(col, k)); //A[j][k] -= A[j][i] * A[i][k];

                }


            }
        }
    }
    else if (_GPU && A->getPos() && P->getPos()) {

        MatrixGPUD matCol(_row, 1, 0, 1);
        initPermMatr << <_numBlocks, _blockSize >> > (P->_matrixGPU, n);
        int k = 0;
        for (int col = 0; col < n; col++) {

            A->getCol(&matCol, col, col);
            double pivotAbs = matCol.max2(&k); // comme matCol est un vecteur colonne, la position du maximum correspond directement à la ligne !
            //double pivot = matCol.get(k, 0, false);


            if (pivotAbs < Tol) {
                //std::cout << "failure, matrix is degenerate" << std::endl;
                throw std::invalid_argument("matrix is degenerate");
            }
            else {
                if (k != col) { //le max pas sur la diagonal
                    //pivoting P and counting pivots starting from N (for determinant)
                    updatePermMatr << <1, 1 >> > (P->_matrixGPU, k, col, n);

                    //pivoting rows of A
                    A->swapLine(col, k);
                }

                updateLUPFactorization <<<n, _blockSize >>> (A->_matrixGPU, col, n);
            }
        }
    }
    else {
        throw std::invalid_argument("Matrix not at the same place");
    }

    // en vrai on peut tout stocker dans une matrice comme on sait que diag(L) = Id, Et donc on peut avoir A = (L-Id) + U


}

void MatrixGPUD::solveSysUpper(MatrixGPUD* U)
{
    if (getNLin() != U->getNCol() || U->getNLin() != getNLin()) {
        throw std::invalid_argument("A must be square");
    }
    int n = getNLin();
    if (!_GPU && !U->getPos()) {

        for (int i = n - 1; i >= 0; i--)
        {
            for (int k = i + 1; k < n; k++) {
                set(i, 0, get(i, 0) - U->get(i, k) * get(k, 0));// x[i] -= A[i][k] * x[k];
            }

            set(i, 0, get(i, 0) / U->get(i, i));
        }
    }
    else if (_GPU && U->getPos()) {
        for (int i = n - 1; i >= 0; i--)
        {
            solveUpSys << < 1, _blockSize >> > (U->_matrixGPU, _matrixGPU, i, n);
        }

    }
    else {
        throw std::invalid_argument("Matrix not at the same place");
    }
}

void MatrixGPUD::solveSysLower(MatrixGPUD* L, MatrixGPUD* b, MatrixGPUD* P) // element diag equal to 1
{
    if (getNLin() != L->getNCol() || L->getNLin() != b->getNLin()) {
        throw std::invalid_argument("A must be square");
    }
    int n = getNLin();
    if (!_GPU && !L->getPos() && !P->getPos() && !b->getPos()) {
        for (int i = 0; i < n; i++) {
            set(i, 0, b->get(P->get(i, 0), 0)); // x[i] = b[P[i]];

            for (int k = 0; k < i; k++) {
                set(i, 0, get(i, 0) - L->get(i, k) * get(k, 0));
            }
        }
    }
    else if (_GPU && L->getPos() && P->getPos() && b->getPos()) {
        setPermute << <_numBlocks, _blockSize >> > (_matrixGPU, b->_matrixGPU, P->_matrixGPU, n);
        for (int i = 0; i < n; i++) {
            solveLowSys << < 1, _blockSize >> > (L->_matrixGPU, _matrixGPU, i, n);
        }
    }
    else {
        throw std::invalid_argument("Matrix not at the same place");
    }

}

void MatrixGPUD::solveSys(MatrixGPUD* A, MatrixGPUD* P, MatrixGPUD* b)
{
    if (A->getNCol() != A->getNLin() || A->getNLin() != b->getNLin()) {
        throw std::invalid_argument("wrong size of A");
    }
    if (b->getNCol() != 1) {
        throw std::invalid_argument("b must be a column vector");
    }
    if (P->getNLin() != (A->getNCol() + 1) || P->getNCol() != 1) {
        throw std::invalid_argument("wrong size of P");
    }

    if (!_GPU && !A->getPos() && !P->getPos() && !b->getPos()) {
       
        int n = getNLin();
       
        for (int i = 0; i < n; i++) {
                set(i, 0, b->get(P->get(i, 0), 0)); // x[i] = b[P[i]];
        }
       
        for (int iter = 0; iter < n; iter++) {
            for (int k = iter + 1; k < n; k++) { // en parallele
                set(k, 0, get(k, 0) - A->get(k, iter) * get(iter, 0)); // x[k] = x[k] - A[k][iter] *x[iter] ; avec k>iter
            }
        }
       
        for (int iter = n - 1; iter >= 0; iter--) {
            set(iter, 0, get(iter, 0) / A->get(iter, iter));
            for (int k = 0; k < iter; k++) { // en parallele
                set(k, 0, get(k, 0) - A->get(k, iter) * get(iter, 0)); // x[k] = x[k] - A[k][iter] *x[iter] ; avec k<iter
            }

        }
  
    }
    else if (_GPU && A->getPos() && P->getPos() && b->getPos()) {
        int n = getNLin();
       
        setPermute << <_numBlocks, _blockSize >> > (_matrixGPU, b->_matrixGPU, P->_matrixGPU, n);
       
        
        solveSysGPU << <1, _blockSize, n * sizeof(double) >> > (A->_matrixGPU, _matrixGPU, n);

    }
    else {
        throw std::invalid_argument("Matrix not at the same place");
    }


}


///////////////////////////////////////////////////////////////////////////////
// Fonction autres
///////////////////////////////////////////////////////////////////////////////

double MatrixGPUD::max2() const
{
    if (_row == 0 || _column == 0) {
        return 0;
        //throw std::out_of_range("Empty Matrix");
    }
    if (!_GPU) {
        double M = fabs(get(0, 0));
        double m = 0;
        for (int i = 0;i < _row;++i)
        {
            for (int j = 0;j < _column;++j)
            {
                m = fabs(get(i, j));
                if (m > M) {
                    M = m;
                }
            }
        }
        return M;
    } 
    else {    
        int numBlocks = _numBlocks;
        unsigned int n = _N;
        double odata = 0;
        double* d_odata;
        if (preallocation) {
            d_odata = _preallocation;
        }
        else {
            std::cout << "allocation !!!" << std::endl;
            hipMalloc((void**)&d_odata, sizeof(double) * numBlocks);
        }
        //std::cout << _numBlocks << std::endl;

        switch (_blockSize) {
        case 512:
            maxMultiBlock<512> << <numBlocks, _blockSize >> > (_matrixGPU, d_odata, n);
            maxMonoBlock<512> << < 1, _blockSize >> > (d_odata, d_odata, numBlocks);
            break;
        case 256:
            maxMultiBlock<256> << <numBlocks, _blockSize >> > (_matrixGPU, d_odata, n);
            maxMonoBlock<256> << < 1, _blockSize >> > (d_odata, d_odata, numBlocks);
            break;
        case 128:
            maxMultiBlock<128> << <numBlocks, _blockSize >> > (_matrixGPU, d_odata, n);
            maxMonoBlock<128> << < 1, _blockSize >> > (d_odata, d_odata, numBlocks);
            break;
        case 64:
            maxMultiBlock< 64> << <numBlocks, _blockSize >> > (_matrixGPU, d_odata, n);
            maxMonoBlock< 64> << < 1, _blockSize >> > (d_odata, d_odata, numBlocks);
            break;
        case 32:
            maxMultiBlock< 32> << <numBlocks, _blockSize >> > (_matrixGPU, d_odata, n);
            maxMonoBlock< 32> << < 1, _blockSize >> > (d_odata, d_odata, numBlocks);
            break;
        case 16:
            maxMultiBlock< 16> << <numBlocks, _blockSize >> > (_matrixGPU, d_odata, n);
            maxMonoBlock< 16> << < 1, _blockSize >> > (d_odata, d_odata, numBlocks);
            break;
        case  8:
            maxMultiBlock<  8> << <numBlocks, _blockSize >> > (_matrixGPU, d_odata, n);
            maxMonoBlock< 8> << < 1, _blockSize >> > (d_odata, d_odata, numBlocks);
            break;
        case  4:
            maxMultiBlock<  4> << <numBlocks, _blockSize >> > (_matrixGPU, d_odata, n);
            maxMonoBlock<  4> << < 1, _blockSize >> > (d_odata, d_odata, numBlocks);
            break;
        case  2:
            maxMultiBlock<  2> << <numBlocks, _blockSize >> > (_matrixGPU, d_odata, n);
            maxMonoBlock<  2> << < 1, _blockSize >> > (d_odata, d_odata, numBlocks);
            break;
        case  1:
            maxMultiBlock<  1> << <numBlocks, _blockSize >> > (_matrixGPU, d_odata, n);
            maxMonoBlock<  1> << < 1, _blockSize >> > (d_odata, d_odata, numBlocks);
            break;
        }
        //hipDeviceSynchronize();
         if (preallocation) {
            hipMemcpy(_preallocationFloat, d_odata, sizeof(double), hipMemcpyDeviceToHost);
            return sqrt(*_preallocationFloat);
        }
        else
        {
            hipMemcpy(&odata, d_odata, sizeof(double), hipMemcpyDeviceToHost);
            std::cout << "free !!!" << std::endl;
            hipFree(d_odata);
            return sqrt(odata);
        }
    }
}

double MatrixGPUD::max2(int* indice)
{
    if (_row == 0 || _column == 0) {
        throw std::out_of_range("Empty Matrix");
    }
    if (!_GPU) {
        double M = fabs(get(0, 0));
        double m = 0;
        for (int i = 0; i < _row; ++i)
        {
            for (int j = 0; j < _column; ++j)
            {
                m = fabs(get(i, j));
                if (m > M) {
                    *indice = i * _column + j;
                    M = m;
                }
            }
        }
        return M;
    }
    else {

        int numBlocks = _numBlocks;
        unsigned int n = _N;
        double odata = 0;
        double* d_odata;
        int* d_pos;
        int pos = 0;
        hipMalloc((void**)&d_pos, sizeof(int) * numBlocks);
        if (preallocation) {

            d_odata = _preallocation;
        }
        else {

            hipMalloc((void**)&d_odata, sizeof(double) * numBlocks);
        }
        //std::cout << _numBlocks << std::endl;

        switch (_blockSize) {
        case 512:
            maxMultiBlock<512> <<<numBlocks, _blockSize >> > (_matrixGPU, d_odata, n, d_pos);
            maxMonoBlock<512> <<< 1, _blockSize >> > (d_odata, d_odata, numBlocks, d_pos);
            break;
        case 256:
            maxMultiBlock<256> << <numBlocks, _blockSize >> > (_matrixGPU, d_odata, n, d_pos);
            maxMonoBlock<256> << < 1, _blockSize >> > (d_odata, d_odata, numBlocks, d_pos);
            break;
        case 128:
            maxMultiBlock<128> << <numBlocks, _blockSize >> > (_matrixGPU, d_odata, n, d_pos);
            maxMonoBlock<128> << < 1, _blockSize >> > (d_odata, d_odata, numBlocks, d_pos);
            break;
        case 64:
            maxMultiBlock< 64> << <numBlocks, _blockSize >> > (_matrixGPU, d_odata, n, d_pos);
            maxMonoBlock< 64> << < 1, _blockSize >> > (d_odata, d_odata, numBlocks, d_pos);
            break;
        case 32:
            maxMultiBlock< 32> << <numBlocks, _blockSize >> > (_matrixGPU, d_odata, n, d_pos);
            maxMonoBlock< 32> << < 1, _blockSize >> > (d_odata, d_odata, numBlocks, d_pos);
            break;
        case 16:
            maxMultiBlock< 16> << <numBlocks, _blockSize >> > (_matrixGPU, d_odata, n, d_pos);
            maxMonoBlock< 16> << < 1, _blockSize >> > (d_odata, d_odata, numBlocks, d_pos);
            break;
        case  8:
            maxMultiBlock<  8> << <numBlocks, _blockSize >> > (_matrixGPU, d_odata, n, d_pos);
            maxMonoBlock< 8> << < 1, _blockSize >> > (d_odata, d_odata, numBlocks, d_pos);
            break;
        case  4:
            maxMultiBlock<  4> << <numBlocks, _blockSize >> > (_matrixGPU, d_odata, n, d_pos);
            maxMonoBlock<  4> << < 1, _blockSize >> > (d_odata, d_odata, numBlocks, d_pos);
            break;
        case  2:
            maxMultiBlock<  2> << <numBlocks, _blockSize >> > (_matrixGPU, d_odata, n, d_pos);
            maxMonoBlock<  2> << < 1, _blockSize >> > (d_odata, d_odata, numBlocks, d_pos);
            break;
        case  1:
            maxMultiBlock<  1> << <numBlocks, _blockSize >> > (_matrixGPU, d_odata, n, d_pos);
            maxMonoBlock<  1> << < 1, _blockSize >> > (d_odata, d_odata, numBlocks, d_pos);
            break;
        }
        //hipDeviceSynchronize();
        hipMemcpy(&odata, d_odata, sizeof(double), hipMemcpyDeviceToHost);
        hipMemcpy(&pos, d_pos, sizeof(int), hipMemcpyDeviceToHost);
        hipFree(d_pos);
        if (!preallocation) {
            hipFree(d_odata);
        }
        *indice = pos;
        return sqrt(odata);
    }
}
double MatrixGPUD::max2(MatrixGPUD* m) const
{
    if (_row == 0 || _column == 0) {
        throw std::out_of_range("Empty Matrix");
    }
    if (!_GPU && !(m->getPos())) 
    {
        double M = fabs(get(0, 0));
        double f = 0;
        for (int i = 0;i < _row;++i)
        {
            for (int j = 0;j < _column;++j)
            {
                f = fabs(get(i, j)- m->get(i,j));
                if (f > M) {
                    M = f;
                }
            }
        }
        return M;
    }
    else if (_GPU && m->getPos()) {
        int numBlocks = _numBlocks;
        unsigned int n = _N;
        double odata;
        double* d_odata;
        if (preallocation) {
            d_odata = _preallocation;
        }
        else {
            std::cout << "allocation !!!" << std::endl;
            hipMalloc((void**)&d_odata, sizeof(double) * numBlocks);
        }
        //std::cout << _numBlocks << std::endl;

        switch (_blockSize) {
        case 512:
            maxMultiBlock<512> << <numBlocks, _blockSize >> > (_matrixGPU, m->_matrixGPU, d_odata, n);
            maxMonoBlock<512> << < 1, _blockSize >> > (d_odata, d_odata, numBlocks);
            break;
        case 256:
            maxMultiBlock<256> << <numBlocks, _blockSize >> > (_matrixGPU, m->_matrixGPU, d_odata, n);
            maxMonoBlock<256> << < 1, _blockSize >> > (d_odata, d_odata, numBlocks);
            break;
        case 128:
            maxMultiBlock<128> << <numBlocks, _blockSize >> > (_matrixGPU, m->_matrixGPU, d_odata, n);
            maxMonoBlock<128> << < 1, _blockSize >> > (d_odata, d_odata, numBlocks);
            break;
        case 64:
            maxMultiBlock< 64> << <numBlocks, _blockSize >> > (_matrixGPU, m->_matrixGPU, d_odata, n);
            maxMonoBlock< 64> << < 1, _blockSize >> > (d_odata, d_odata, numBlocks);
            break;
        case 32:
            maxMultiBlock< 32> << <numBlocks, _blockSize >> > (_matrixGPU, m->_matrixGPU, d_odata, n);
            maxMonoBlock< 32> << < 1, _blockSize >> > (d_odata, d_odata, numBlocks);
            break;
        case 16:
            maxMultiBlock< 16> << <numBlocks, _blockSize >> > (_matrixGPU, m->_matrixGPU, d_odata, n);
            maxMonoBlock< 16> << < 1, _blockSize >> > (d_odata, d_odata, numBlocks);
            break;
        case  8:
            maxMultiBlock<  8> << <numBlocks, _blockSize >> > (_matrixGPU, m->_matrixGPU, d_odata, n);
            maxMonoBlock< 8> << < 1, _blockSize >> > (d_odata, d_odata, numBlocks);
            break;
        case  4:
            maxMultiBlock<  4> << <numBlocks, _blockSize >> > (_matrixGPU, m->_matrixGPU, d_odata, n);
            maxMonoBlock<  4> << < 1, _blockSize >> > (d_odata, d_odata, numBlocks);
            break;
        case  2:
            maxMultiBlock<  2> << <numBlocks, _blockSize >> > (_matrixGPU, m->_matrixGPU, d_odata, n);
            maxMonoBlock<  2> << < 1, _blockSize >> > (d_odata, d_odata, numBlocks);
            break;
        case  1:
            maxMultiBlock<  1> << <numBlocks, _blockSize >> > (_matrixGPU, m->_matrixGPU, d_odata, n);
            maxMonoBlock<  1> << < 1, _blockSize >> > (d_odata, d_odata, numBlocks);
            break;
        }
        //hipDeviceSynchronize();
        if (preallocation) {
            hipMemcpy(_preallocationFloat, d_odata, sizeof(double), hipMemcpyDeviceToHost);
            return sqrt(*_preallocationFloat);
        }
        else
        {
            hipMemcpy(&odata, d_odata, sizeof(double), hipMemcpyDeviceToHost);
            std::cout << "free !!!" << std::endl;
            hipFree(d_odata);
            return sqrt(odata);
        }
    }
    else {
        throw std::invalid_argument("Matrix not at the same place");
    }
}

double MatrixGPUD::distance2(MatrixGPUD* m)
{
    if (!dim(m)) {
        throw std::invalid_argument("not the same size");
    }
    if (_GPU && m->getPos())
    {
        int numBlocks = _numBlocks;
        unsigned int n = _N;
        double odata = 0;
        double* d_odata;
        if (preallocation) {
            d_odata = _preallocation;
        }
        else {
            hipMalloc((void**)&d_odata, sizeof(double) * numBlocks);
        }
        //std::cout << _numBlocks << std::endl;

        switch (_blockSize) {
        case 512:
            distanceMultiBlock<512> << <numBlocks, _blockSize >> > (_matrixGPU, m->_matrixGPU, d_odata, n);
            sumMonoBlock<512> << < 1, _blockSize >> > (d_odata, d_odata, numBlocks);
            break;
        case 256:
            distanceMultiBlock<256> << <numBlocks, _blockSize >> > (_matrixGPU, m->_matrixGPU, d_odata, n);
            sumMonoBlock<256> << < 1, _blockSize >> > (d_odata, d_odata, numBlocks);
            break;
        case 128:
            distanceMultiBlock<128> << <numBlocks, _blockSize >> > (_matrixGPU, m->_matrixGPU, d_odata, n);
            sumMonoBlock<128> << < 1, _blockSize >> > (d_odata, d_odata, numBlocks);
            break;
        case 64:
            distanceMultiBlock< 64> << <numBlocks, _blockSize >> > (_matrixGPU, m->_matrixGPU, d_odata, n);
            sumMonoBlock< 64> << < 1, _blockSize >> > (d_odata, d_odata, numBlocks);
            break;
        case 32:
            distanceMultiBlock< 32> << <numBlocks, _blockSize >> > (_matrixGPU, m->_matrixGPU, d_odata, n);
            sumMonoBlock< 32> << < 1, _blockSize >> > (d_odata, d_odata, numBlocks);
            break;
        case 16:
            distanceMultiBlock< 16> << <numBlocks, _blockSize >> > (_matrixGPU, m->_matrixGPU, d_odata, n);
            sumMonoBlock< 16> << < 1, _blockSize >> > (d_odata, d_odata, numBlocks);
            break;
        case  8:
            distanceMultiBlock<  8> << <numBlocks, _blockSize >> > (_matrixGPU, m->_matrixGPU, d_odata, n);
            sumMonoBlock< 8> << < 1, _blockSize >> > (d_odata, d_odata, numBlocks);
            break;
        case  4:
            distanceMultiBlock<  4> << <numBlocks, _blockSize >> > (_matrixGPU, m->_matrixGPU, d_odata, n);
            sumMonoBlock<  4> << < 1, _blockSize >> > (d_odata, d_odata, numBlocks);
            break;
        case  2:
            distanceMultiBlock<  2> << <numBlocks, _blockSize >> > (_matrixGPU, m->_matrixGPU, d_odata, n);
            sumMonoBlock<  2> << < 1, _blockSize >> > (d_odata, d_odata, numBlocks);
            break;
        case  1:
            distanceMultiBlock<  1> << <numBlocks, _blockSize >> > (_matrixGPU, m->_matrixGPU, d_odata, n);
            sumMonoBlock<  1> << < 1, _blockSize >> > (d_odata, d_odata, numBlocks);
            break;
        }
        //hipDeviceSynchronize();
        hipMemcpy(&odata, d_odata, sizeof(double), hipMemcpyDeviceToHost);
        if (!preallocation) {
            hipFree(d_odata);
        }


        return sqrtf(odata);
    }
    else if (!_GPU && !(m->getPos()))
    {
        double d = 0;
        double r = 0;
        for (int i = 0;i < _row;++i)
        {
            for (int j = 0;j < _column;++j)
            {
                r = get(i, j) - m->get(i, j);
                d = d + r * r;
            }
        }
        return sqrtf(d);
    }
    else {
        throw std::invalid_argument("Matrix not at the same place");
    }
}

void MatrixGPUD::Moy(MatrixGPUD* m, MatrixGPUD* nb, int sens)
{
    double s;
    int n;
    if (sens) { // on travaille sur les colonnes
        if ((_row != 1) || (_column != m->getNCol()) || (_column != nb->getNCol()) || (nb->getNLin() != 1))
        {
            throw std::invalid_argument("wrong dimension of the vector");
        }
        if (_GPU && nb->getPos() && m->getPos())
        {
            
            moyGPU1 <<<_numBlocks, _blockSize >>> (_matrixGPU, m->_matrixGPU, nb->_matrixGPU, m->getNLin(), _column);
        }
        else if ((!_GPU) && !(nb->getPos()) && !(m->getPos())) {
            for (int j = 0; j < _column;j++)
            {
                n = nb->get(0, j);
                s = 0;
                if (n > 0)
                {
                    for (int i = 0; i < m->getNLin();i++)
                    {
                        s = s + m->get(i, j);
                    }
                    s = s / n;
                }
                set(0, j, s);
            }
        }
        else {
            throw std::invalid_argument("Matrix not at the same place");
        }
        

    }
    else { // on travaille sur les lignes 
        if ((_column != 1) || (_row != m->getNLin()) || (_row != nb->getNLin()) || (nb->getNCol() != 1)) {
            throw std::invalid_argument("wrong dimension of the vector");
        }
        if (_GPU && nb->getPos())
        {
            
            moyGPU2<<<_numBlocks, _blockSize >> > (_matrixGPU, m->_matrixGPU, nb->_matrixGPU, _row , m->getNCol());
        }
        else if ((!_GPU) && !(nb->getPos())) {
            for (int i = 0; i < _row;i++)
            {
                n = nb->get(i, 0);
                s = 0;
                if (n > 0) {
                    for (int j = 0; j < m->getNCol();j++)
                    {
                        s = s + m->get(i, j);
                    }
                    s = s / n;
                }
                set(i, 0, s);
            }
        }
        else {
            throw std::invalid_argument("Matrix not at the same place");
        }

    }
}

void MatrixGPUD::project(MatrixGPUD* Lb, MatrixGPUD* Ub)
{
    if (!dim(Lb) || !dim(Ub)) {
        throw std::invalid_argument("not the same dimension");
    }
    if (_GPU && Lb->getPos() && Ub->getPos())
    {
        projectGPU<<<_numBlocks, _blockSize >>>(_matrixGPU, Lb->_matrixGPU, Ub->_matrixGPU, _N);
    }
    else if (!_GPU && !(Lb->getPos()) && !(Ub->getPos()))
    {
        double ub = 0;
        double lb = 0;
        double r = 0;
        MatrixGPUD temp(*this);
        for (int i = 0;i < _row;++i)
        {
            for (int j = 0;j < _column;++j)
            {
                r = get(i, j);
                ub = Ub->get(i, j);
                lb = Lb->get(i, j);
                if (ub < lb) {
                    throw std::invalid_argument("impossible to have a value for the projection, ub>lb");
                }
                r = (ub - r) * (r > ub) + (lb - r) * (r < lb) + r; // permet de ne pas faire de branchement if.
                temp.set(i, j, r);
            }
        }
        this->set(&temp);
    }
    else {
        throw std::invalid_argument("Matrix not at the same place");
    }
    
}
void MatrixGPUD::projectNeg()
{
    if (_GPU)
    {
        projectGPUNeg << <_numBlocks, _blockSize >> > (_matrixGPU, _N);
    }
    else if (!_GPU)
    {
        for (int i = 0; i < _row; ++i)
        {
            for (int j = 0; j < _column; ++j)
            {
                double r = get(i, j);
                r = (r > 0) * r;
                set(i, j, r);
            }
        }
    }

}
void MatrixGPUD::projectPos()
{
   
    if (_GPU)
    {
        projectGPUPos <<<_numBlocks, _blockSize >> > (_matrixGPU, _N);
    }
    else if (!_GPU)
    {
        for (int i = 0; i < _row; ++i)
        {
            for (int j = 0; j < _column; ++j)
            {
                double r = get(i, j);
                r = (r > 0) * r;
                set(i, j, r);
            }
        }
    }
}



double MatrixGPUD::sum() const
{
    if (_row == 0 || _column == 0) {
        return 0;
        //throw std::out_of_range("Empty Matrix");
    }
    if (_GPU) 
    {
        int numBlocks = _numBlocks;
        unsigned int n = _N;
        double odata = 0;
        double* d_odata;
        if (preallocation) {
            d_odata = _preallocation;
        }
        else {
            hipMalloc((void**)&d_odata, sizeof(double) * numBlocks);
        }


        switch (_blockSize) {
        case 512:
            SumMultiBlock<512> <<<numBlocks, _blockSize >> > (_matrixGPU, d_odata, n);
            sumMonoBlock<512> <<< 1, _blockSize >> > (d_odata, d_odata, numBlocks);
            break;
        case 256:
            SumMultiBlock<256> << <numBlocks, _blockSize >> > (_matrixGPU, d_odata, n);
            sumMonoBlock<256> << < 1, _blockSize >> > (d_odata, d_odata, numBlocks);
            break;
        case 128:
            SumMultiBlock<128> << <numBlocks, _blockSize >> > (_matrixGPU, d_odata, n);
            sumMonoBlock<128> << < 1, _blockSize >> > (d_odata, d_odata, numBlocks);
            break;
        case 64:
            SumMultiBlock< 64> << <numBlocks, _blockSize >> > (_matrixGPU, d_odata, n);
            sumMonoBlock< 64> << < 1, _blockSize >> > (d_odata, d_odata, numBlocks);
            break;
        case 32:
            SumMultiBlock< 32> << <numBlocks, _blockSize >> > (_matrixGPU, d_odata, n);
            sumMonoBlock< 32> << < 1, _blockSize >> > (d_odata, d_odata, numBlocks);
            break;
        case 16:
            SumMultiBlock< 16> << <numBlocks, _blockSize >> > (_matrixGPU, d_odata, n);
            sumMonoBlock< 16> << < 1, _blockSize >> > (d_odata, d_odata, numBlocks);
            break;
        case  8:
            SumMultiBlock<  8> << <numBlocks, _blockSize >> > (_matrixGPU, d_odata, n);
            sumMonoBlock< 8> << < 1, _blockSize >> > (d_odata, d_odata, numBlocks);
            break;
        case  4:
            SumMultiBlock<  4> << <numBlocks, _blockSize >> > (_matrixGPU, d_odata, n);
            sumMonoBlock<  4> << < 1, _blockSize >> > (d_odata, d_odata, numBlocks);
            break;
        case  2:
            SumMultiBlock<  2> << <numBlocks, _blockSize >> > (_matrixGPU, d_odata, n);
            sumMonoBlock<  2> << < 1, _blockSize >> > (d_odata, d_odata, numBlocks);
            break;
        case  1:
            SumMultiBlock<  1> << <numBlocks, _blockSize >> > (_matrixGPU, d_odata, n);
            sumMonoBlock<  1> << < 1, _blockSize >> > (d_odata, d_odata, numBlocks);
            break;
        }
        //hipDeviceSynchronize();
        hipMemcpy(&odata, d_odata, sizeof(double), hipMemcpyDeviceToHost);
        if (!preallocation) {
            hipFree(d_odata);
        }
        //std::cout << "sum " << odata << " " <<_blockSize << " " << numBlocks << std::endl;
        return odata;
    }
    else 
    {
        double d = 0;
        double r = 0;
        for (int i = 0;i < _row;++i)
        {
            for (int j = 0;j < _column;++j)
            {
                r = get(i, j);
                d = d + r;
            }
        }
        return d;
    }
}

double MatrixGPUD::sum(int begin, int end)
{
    if (begin < 0 || end < 0) {
        throw std::invalid_argument("indice must be positve");
    }
    if (begin > end) {
        throw std::invalid_argument("begin must be smaller than end");
    }
    if (begin > _N || end > _N) {
        throw std::out_of_range("indice must smaller than N");
    }
    if (_row == 0 || _column == 0) {
        return 0;
        //throw std::out_of_range("Empty Matrix");
    }

    if (_GPU)
    {
        int numBlocks = _numBlocks;
        double odata = 0;
        double* d_odata;
        if (preallocation) {
            d_odata = _preallocation;
        }
        else {
            hipMalloc((void**)&d_odata, sizeof(double) * numBlocks);
        }


        switch (_blockSize) {
        case 512:
            SumMultiBlock<512> << <numBlocks, _blockSize >> > (_matrixGPU, d_odata, begin, end);
            sumMonoBlock<512> << < 1, _blockSize >> > (d_odata, d_odata, numBlocks);
            break;
        case 256:
            SumMultiBlock<256> << <numBlocks, _blockSize >> > (_matrixGPU, d_odata, begin, end);
            sumMonoBlock<256> << < 1, _blockSize >> > (d_odata, d_odata, numBlocks);
            break;
        case 128:
            SumMultiBlock<128> << <numBlocks, _blockSize >> > (_matrixGPU, d_odata, begin, end);
            sumMonoBlock<128> << < 1, _blockSize >> > (d_odata, d_odata, numBlocks);
            break;
        case 64:
            SumMultiBlock< 64> << <numBlocks, _blockSize >> > (_matrixGPU, d_odata, begin, end);
            sumMonoBlock< 64> << < 1, _blockSize >> > (d_odata, d_odata, numBlocks);
            break;
        case 32:
            SumMultiBlock< 32> << <numBlocks, _blockSize >> > (_matrixGPU, d_odata, begin, end);
            sumMonoBlock< 32> << < 1, _blockSize >> > (d_odata, d_odata, numBlocks);
            break;
        case 16:
            SumMultiBlock< 16> << <numBlocks, _blockSize >> > (_matrixGPU, d_odata, begin, end);
            sumMonoBlock< 16> << < 1, _blockSize >> > (d_odata, d_odata, numBlocks);
            break;
        case  8:
            SumMultiBlock<  8> << <numBlocks, _blockSize >> > (_matrixGPU, d_odata, begin, end);
            sumMonoBlock< 8> << < 1, _blockSize >> > (d_odata, d_odata, numBlocks);
            break;
        case  4:
            SumMultiBlock<  4> << <numBlocks, _blockSize >> > (_matrixGPU, d_odata, begin, end);
            sumMonoBlock<  4> << < 1, _blockSize >> > (d_odata, d_odata, numBlocks);
            break;
        case  2:
            SumMultiBlock<  2> << <numBlocks, _blockSize >> > (_matrixGPU, d_odata, begin, end);
            sumMonoBlock<  2> << < 1, _blockSize >> > (d_odata, d_odata, numBlocks);
            break;
        case  1:
            SumMultiBlock<  1> << <numBlocks, _blockSize >> > (_matrixGPU, d_odata, begin, end);
            sumMonoBlock<  1> << < 1, _blockSize >> > (d_odata, d_odata, numBlocks);
            break;
        }
        //hipDeviceSynchronize();
        hipMemcpy(&odata, d_odata, sizeof(double), hipMemcpyDeviceToHost);
        if (!preallocation) {
            hipFree(d_odata);
        }
        //std::cout << "sum " << odata << " " <<_blockSize << " " << numBlocks << std::endl;
        return odata;
    }
    else if (!_GPU)
    {
        double d = 0;
        double r = 0;
        for (int elem = begin; elem < end; ++elem)
        {

            r = _matrixCPU[elem];
            d = d + r;
        }
        return d;
    }
}

void MatrixGPUD::sum(MatrixGPUD* m)
{
    double s = 0;
     // on travaille sur les lignes 
    if ((_column != 1) || (_row != m->getNLin())) {
        throw std::invalid_argument("wrong dimension of the column vector ");
    }
    int col = m->getNCol();
    if (_GPU && m->getPos())
    {
        int numBlocks = _row;
        switch (_blockSize) {
        case 512:
            SumEachRow<512> << <numBlocks, _blockSize >> > (m->_matrixGPU, _matrixGPU, col);
            break;
        case 256:
            SumEachRow<256> << <numBlocks, _blockSize >> > (m->_matrixGPU, _matrixGPU, col);
            break;
        case 128:
            SumEachRow<128> << <numBlocks, _blockSize >> > (m->_matrixGPU, _matrixGPU, col);
            break;
        case 64:
            SumEachRow< 64> << <numBlocks, _blockSize >> > (m->_matrixGPU, _matrixGPU, col);
            break;
        case 32:
            SumEachRow< 32> << <numBlocks, _blockSize >> > (m->_matrixGPU, _matrixGPU, col);
            break;
        case 16:
            SumEachRow< 16> << <numBlocks, _blockSize >> > (m->_matrixGPU, _matrixGPU, col);
            break;
        case  8:
            SumEachRow<  8> << <numBlocks, _blockSize >> > (m->_matrixGPU, _matrixGPU, col);
            break;
        case  4:
            SumEachRow<  4> << <numBlocks, _blockSize >> > (m->_matrixGPU, _matrixGPU, col);
            break;
        case  2:
            SumEachRow<  2> << <numBlocks, _blockSize >> > (m->_matrixGPU, _matrixGPU, col);
            break;
        case  1:
            SumEachRow<  1> << <numBlocks, _blockSize >> > (m->_matrixGPU, _matrixGPU, col);
            break;
        }
    }
    else if (!_GPU && !(m->getPos()))
    {
        for (int i = 0; i < _row;i++)
        {
            s = 0;
            for (int j = 0; j < col;j++)
            {
                s = s + m->get(i, j);
            }
            set(i, 0, s);
        }
    }
    else {
        throw std::invalid_argument("Matrix not at the same place");
    }
}

double MatrixGPUD::distance2() {

    if (_GPU ) //&& m->getPos())
    {
        int numBlocks = _numBlocks;
        unsigned int n = _N;
        double* d_odata;
        if (preallocation) {
            d_odata = _preallocation;
        }
        else {
            hipMalloc((void**)&d_odata, sizeof(double) * numBlocks);
        }
        double odata = 0;
        
        
        //std::cout << _numBlocks << std::endl;
 
        switch (_blockSize) {
        case 512:
            distanceMultiBlock<512> << <numBlocks, _blockSize >> > (_matrixGPU, d_odata, n);
            sumMonoBlock<512> << < 1, _blockSize >> > (d_odata, d_odata, numBlocks);
        break;
        case 256:
            distanceMultiBlock<256> <<<numBlocks, _blockSize >> > (_matrixGPU, d_odata, n);
            sumMonoBlock<256> <<< 1, _blockSize >> > (d_odata, d_odata, numBlocks);
        break;
        case 128:
            distanceMultiBlock<128> <<<numBlocks, _blockSize >> > (_matrixGPU, d_odata, n);
            sumMonoBlock<128> <<< 1, _blockSize >> > (d_odata, d_odata, numBlocks);
        break;
        case 64:
            distanceMultiBlock< 64> <<<numBlocks, _blockSize >> > (_matrixGPU, d_odata, n);
            sumMonoBlock< 64> <<< 1, _blockSize >> > (d_odata, d_odata, numBlocks);
            break;
        case 32:
            distanceMultiBlock< 32> <<<numBlocks, _blockSize >> > (_matrixGPU, d_odata, n);
            sumMonoBlock< 32> <<< 1, _blockSize >> > (d_odata, d_odata, numBlocks);
            break;
        case 16:
            distanceMultiBlock< 16> <<<numBlocks, _blockSize >> > (_matrixGPU, d_odata, n);
            sumMonoBlock< 16> <<< 1, _blockSize >> > (d_odata, d_odata, numBlocks);
            break;
        case  8:
            distanceMultiBlock<  8> <<<numBlocks, _blockSize >> > (_matrixGPU, d_odata, n);
            sumMonoBlock< 8> <<< 1, _blockSize >> > (d_odata, d_odata, numBlocks);
            break;
        case  4:
            distanceMultiBlock<  4> <<<numBlocks, _blockSize >> > (_matrixGPU, d_odata, n);
            sumMonoBlock<  4> <<< 1, _blockSize >> > (d_odata, d_odata, numBlocks);
            break;
        case  2:
            distanceMultiBlock<  2> <<<numBlocks, _blockSize >> > (_matrixGPU, d_odata, n);
            sumMonoBlock<  2> <<< 1, _blockSize >> > (d_odata, d_odata, numBlocks);
            break;
        case  1:
            distanceMultiBlock<  1> <<<numBlocks, _blockSize >>> (_matrixGPU, d_odata, n);
            sumMonoBlock<  1> <<< 1, _blockSize >> > (d_odata, d_odata, numBlocks);
            break; 
        }
        //hipDeviceSynchronize();
        hipMemcpy(&odata, d_odata, sizeof(double), hipMemcpyDeviceToHost);
        if (!preallocation) {
            hipFree(d_odata);
        }
        return sqrtf(odata);
    }
    else if (!_GPU)// && !(m->getPos()))
    {
        double d = 0;
        double r = 0;
        for (int i = 0;i < _row;++i)
        {
            for (int j = 0;j < _column;++j)
            {
                r = get(i, j);// -m->get(i, j);
                d = d + r * r;
            }
        }
        return sqrtf(d); 
    }
}



///////////////////////////////////////////////////////////////////////////////
// Display MatrixGPUD contents
///////////////////////////////////////////////////////////////////////////////
void MatrixGPUD::display(bool force) 
{   
    bool transfert = false;
    if (this) {
        if (_GPU && !force ) {
           std::cout << " Matrix stockee sur GPU, faire le transfertCPU avant d'afficher " << std::endl;
        }
        if (_row == 0 || _column == 0)
        {
            std::cout << "matrix vide " << std::endl;
            return;
        }
        else {
            
            if (_GPU) {
                transferCPU();
                transfert = true;
            }
            if (_column == 1) {
                std::cout << " transpose  : ";
                for (int i = 0;i < _row;++i)
                {
                    for (int j = 0;j < _column;++j)
                    {
                        double value = get(i, j);
                        std::cout << std::setprecision(7) << value;
                        //std::cout << std::fixed << std::setprecision(2) << value;
                        std::cout << " ";
                    }
                }
                std::cout << std::endl;
            }
            else {
                for (int i = 0;i < _row;++i)
                {
                    for (int j = 0;j < _column;++j)
                    {
                        double value = get(i, j);
                        std::cout << std::setprecision(7) << value;
                        //std::cout << std::fixed << std::setprecision(3) << value;
                        if (j != _column - 1) std::cout << " ";
                    }

                    std::cout << std::endl;
                }
                std::cout << std::endl;
            }
            if (transfert) {
                transferGPU();
            }
        }
    }
    else 
    {
        std::cout << "matrix non definie " << std::endl;
    }
}

void MatrixGPUD::displayBloc(int iBegin, int iEnd, int jBegin, int jEnd, bool force)
{
    if ((iBegin < 0) || (jBegin < 0) || iEnd > _row || jEnd > _column) {
        throw std::out_of_range("index out of bounds");
    } if ((iBegin > iEnd) || (jBegin > jEnd)) {
        throw std::invalid_argument("xBegin must be smaller than xEnd");
    }
    bool transfert = false;
    if (this) {
        if (_GPU && !force) {
            std::cout << " Matrix stockee sur GPU, faire le transfertCPU avant d'afficher " << std::endl;
        }
        else {
            if (_row == 0 || _column == 0)
            {
                std::cout << "matrix vide " << std::endl;
            }
            if (_GPU) {
                transferCPU();
                transfert = true;
            }
            if (jEnd - jBegin == 1 ) {
                std::cout << " transpose  : ";
                for (int i = iBegin; i < iEnd; ++i)
                {
                    double value = get(i, jBegin);
                    std::cout << std::setprecision(7) << value;
                    std::cout << " ";
                }
                std::cout << std::endl;
            }
            else {
                for (int i = iBegin; i < iEnd; ++i)
                {
                    for (int j = jBegin; j < jEnd; ++j)
                    {
                        double value = get(i, j);
                        std::cout << std::setprecision(7) << value;
                        if (j != jEnd - 1) {
                            std::cout << " ";
                        }
                    }

                    std::cout << std::endl;
                }
                std::cout << std::endl;
            }
            if (transfert) {
                transferGPU();
            }
        }
    }
    else
    {
        std::cout << "matrix non definie " << std::endl;
    }
}

void MatrixGPUD::swapLine(int line1, int line2)
{
    if (_GPU) {
        swapLineGJ << <_numBlocks, _blockSize >> > (_matrixGPU, line1, line2, _column);// swap des lignes
    }
    else {
        double temp = 0;
        for (int i = 0; i < _column; i++) {
            temp = get(line1, i);
            set(line1, i, get(line2, i));
            set(line2, i, temp);
        }
    }
    
}




///////////////////////////////////////////////////////////////////////////////
// Destructor
///////////////////////////////////////////////////////////////////////////////
MatrixGPUD::~MatrixGPUD()
{
    #ifdef DEBUG_DESTRUCTOR
        std::cout << "destruction matrix " << _matrixGPU << std::endl;
    #endif // DEBUG_DESTRUCTOR
    if (_preallocationFloat != nullptr) {
        hipHostFree(_preallocationFloat);
        _preallocationFloat = nullptr;
    }
    if (_preallocation != nullptr) {
        hipFree(_preallocation);
        _preallocation = nullptr;
    }
    if (_matrixGPU) {
        hipFree(_matrixGPU);
        _matrixGPU = nullptr;
    }
     DELETEA(_matrixCPU);
    
}



void MatrixGPUD::saveCSV(const std::string& filename, std::ios_base::openmode mode, int trans) const
{
    if (_GPU) {
        throw std::domain_error("Matrix on GPU");
    }
    std::ofstream myfile;
    myfile.open(filename, mode);
    myfile.precision(50);
    if (!trans) {
        for (int i = 0; i < _row; i++) {
            for (int j = 0; j < _column;j++) {
                myfile << get(i, j) << ";";
            }
            myfile << "\n";
        }
    }
    else {
        for (int j = 0; j < _column;j++) {
            for (int i = 0; i < _row; i++) {
                myfile << get(i, j) << ";";
            }
            myfile << "\n";
        }
    }

    myfile.close();
}

void MatrixGPUD::saveCSVForce(const std::string& filename, std::ios_base::openmode mode, int trans)
{
    int transfert = 0;
    if (_GPU) {
        transfert = 1;
        transferCPU();
    }
    std::ofstream myfile;
    myfile.open(filename, mode);
    myfile.precision(10);
    if (!trans) {
        for (int i = 0; i < _row; i++) {
            for (int j = 0; j < _column; j++) {
                myfile << get(i, j) << ";";
            }
            myfile << "\n";
        }
    }
    else {
        for (int j = 0; j < _column; j++) {
            for (int i = 0; i < _row; i++) {
                myfile << get(i, j) << ";";
            }
            myfile << "\n";
        }
    }

    myfile.close();
    if (transfert) {
        transferGPU();
    }
}


///////////////////////////////////////////////////////////////////////////////
// Fonction globale
///////////////////////////////////////////////////////////////////////////////


__global__ void setup_kernelD(hiprandState* state) {

    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    hiprand_init(1234, idx, 0, &state[idx]);
}


__global__ void generate_kernel(hiprandState* my_curandstate, double* result, double eps, const unsigned int N) {

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int step = blockDim.x * gridDim.x;
    for (int i = index; i < N; i += step)
    {
        result[i] = (2*hiprand_uniform(my_curandstate + i)-1) * eps;
    }
}





__global__ void setGPU(double* mat1, double* mat2, int N) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int step = blockDim.x * gridDim.x;
    for (int i = index; i < N; i += step)
    {
        mat1[i] = mat2[i];
    }
}

__global__ void setGPU(double* mat1, const double value, int N) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int step = blockDim.x * gridDim.x;
    for (int i = index; i < N; i += step)
    {
        mat1[i] = value;
    }
}
__global__ void setGPUunique(double* mat1, const double value, int pos) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int step = blockDim.x * gridDim.x;
    if (index == 0) {
        mat1[pos] = value;
    }

}

__global__ void setTransGPU(double* mat1, double* matToTrans, const int column, const int row) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int step = blockDim.x * gridDim.x;
    const int N = column * row;

    for (int e = index; e < N; e += step)
    {
        int i = e / column;
        int j = e % column;
        mat1[e] = matToTrans[j * row + i];

    }
}

__global__ void setColGPU(double* mat1, double* mat2, const int numCol, const int column, const int row, const int offset) {
    
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int step = blockDim.x * gridDim.x;
    for (int i = index; i < row; i += step)
    {
        mat1[i] = i < offset ? 0 : mat2[i*column+numCol];
    }

}

__global__ void setEyesGPU(double* mat2, const double value, const int col, const int row) 
{
    int N = row * col;
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int step = blockDim.x * gridDim.x;

    for (int l = index; l < N; l+=step) {
        int i = l / col;
        int j = l % col;
        mat2[l] = (i==j) ? value : 0; // pas coalescent, mais bon...
    }
}
__global__ void setEyesGPU(double* mat2, double* mat1, const int col, const int row)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int step = blockDim.x * gridDim.x;
    int N = row * col;
    for (int l = index; l < N; l += step) {
        int i = l / col;
        int j = l % col;
        mat2[l] = (i == j) ? mat1[i] : 0; // pas coalescent, mais bon...
    }
}


__global__ void SetBlocGPU(double* out, double* in, int ibegin, int iend, int jbegin, int jend, int col)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int step = blockDim.x * gridDim.x;
    int offset = jbegin + ibegin * col;
    int N = (jend - jbegin) * (iend - ibegin);

    for (int j = index; j < N; j += step)
    {
        int rowLoc = j / (jend - jbegin);
        int colLoc = j % (jend - jbegin);
        int GlobalInd = offset + rowLoc *col + colLoc;
        out[GlobalInd] = in[j];
    }
}

__global__ void SetBlocGPU(double* out, double* in, int ibegin, int iend, int jbegin, int jend, int col, double factor)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int step = blockDim.x * gridDim.x;
    int offset = jbegin + ibegin * col;
    int N = (jend - jbegin) * (iend - ibegin);

    for (int j = index; j < N; j += step)
    {
        int rowLoc = j / (jend - jbegin);
        int colLoc = j % (jend - jbegin);
        int GlobalInd = offset + rowLoc * col + colLoc;
        out[GlobalInd] = factor * in[j];
    }
}

/*__global__ void SetBlocGPU(double* out, double* in, int ibegin, int iend, int jbegin, int jend, int col, double factor) // fait que la première ligne
{
    int indexX = threadIdx.x + blockIdx.x * blockDim.x;
    int stepX = blockDim.x * gridDim.x;
    int indexY = threadIdx.y + blockIdx.y * blockDim.y;
    int stepY = blockDim.y * gridDim.y;

    for (int j = indexX + jbegin; j < jend; j += stepX)
    {
        for (int i = indexY + ibegin; i < iend; i += stepY)
        {
            out[j + i * col] = factor * in[indexX + indexY * col];
        }
    }
}*/




__global__ void replaceGPU(double* mat,const double previous, const double newValue,const int N) 
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int step = blockDim.x * gridDim.x;
    for (int i = index; i < N; i += step)
    {
        mat[i] = (mat[i] == previous) * (newValue-mat[i]) + mat[i];
    }
}




__global__ void addGPU(double* mat, double c, int N) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int step = blockDim.x * gridDim.x;
    for (int i = index; i < N; i += step)
    {
        mat[i] = mat[i] + c;
    }
}
__global__ void addGPU(double* mat1, double* mat2, double c, int N) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int step = blockDim.x * gridDim.x;
    for (int i = index; i < N; i += step)
    {
        mat1[i] = mat2[i] + c;
    }
}
__global__ void addGPU(double* mat1, double* mat2, int N) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int step = blockDim.x * gridDim.x;
    for (int i = index; i < N; i += step)
    {
        mat1[i] = mat1[i] + mat2[i];
    }
}
__global__ void addGPU(double* mat1, double* mat2, double* mat3, int N) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int step = blockDim.x * gridDim.x;
    for (int i = index; i < N; i += step)
    {
        mat1[i] = mat2[i] + mat3[i];
    }
}

__global__ void addVectorGPU1(double* mat1, double* vect, const int n, int N) //vecteur colonne
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int step = blockDim.x * gridDim.x;
    for (int i = index; i < N; i += step)
    {
        int k = i / n; // division entière
        mat1[i] = mat1[i] + vect[k];
    }

}
__global__ void addVectorGPU2(double* mat1, double* vect, const int n, int N) // vecteur ligne
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int step = blockDim.x * gridDim.x;
    for (int i = index; i < N; i += step)
    {
        int k = i % n; // modulo
        mat1[i] = mat1[i] + vect[k];
    }


}

__global__ void addTransGPU(double* out, double* mat1, double* mat2, const int col, const int line, int N) 
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int step = blockDim.x * gridDim.x;
    for (int l = index; l < N; l += step)
    {
        int i = l / col;
        int j = l % col;
        int k = i + j * line;
        out[l] = mat1[l] + mat2[k];
    }
}

__global__ void substractGPU(double* mat1, double* mat2, int N) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int step = blockDim.x * gridDim.x;
    for (int i = index; i < N; i += step)
    {
        mat1[i] = mat1[i] - mat2[i];
    }
}
__global__ void substractGPU(double* mat1, double* mat2, double* mat3, int N) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int step = blockDim.x * gridDim.x;
    for (int i = index; i < N; i += step)
    {
        mat1[i] = mat2[i] - mat3[i];
    }
}

__global__ void substractVectorGPU1(double* mat1, double* vect, const int n, int N) //vecteur colonne
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int step = blockDim.x * gridDim.x;
    for (int i = index; i < N; i += step)
    {
        int k = i / n; // division entière
        mat1[i] = mat1[i] - vect[k];
    }

}
__global__ void substractVectorGPU2(double* mat1, double* vect, const int n, int N) // vecteur ligne
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int step = blockDim.x * gridDim.x;
    for (int i = index; i < N; i += step)
    {
        int k = i % n; // modulo
        mat1[i] = mat1[i] - vect[k];
    }

}

__global__ void substractTransGPU(double* out, double* mat1, double* mat2, const int col, const int line, int N)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int step = blockDim.x * gridDim.x;
    for (int l = index; l < N; l += step)
    {
        int i = l / col;
        int j = l % col;
        int k = i + j * line;
        out[l] = mat1[l] - mat2[k];
    }
}

__global__ void multiplyGPU(double* mat, const double c, int N) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int step = blockDim.x * gridDim.x;
    for (int i = index; i < N; i += step)
    {
        mat[i] = mat[i] * c;
    }
}

__global__ void multiplyTGPU(double* mat1, double* mat2, int N) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int step = blockDim.x * gridDim.x;
    for (int i = index; i < N; i += step)
    {
        mat1[i] = mat1[i] * mat2[i];
    }
}
__global__ void multiplyTGPU(double* mat1, double* mat2, double* mat3, int N) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int step = blockDim.x * gridDim.x;
    for (int i = index; i < N; i += step)
    {
        mat1[i] = mat2[i] * mat3[i];
    }
}

__global__ void divideGPU(double* mat, const double c, int N) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int step = blockDim.x * gridDim.x;
    for (int i = index; i < N; i += step)
    {
        mat[i] = mat[i] / c;
    }
}
__global__ void divideGPU(double* mat1, double* mat2, int N) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int step = blockDim.x * gridDim.x;
    for (int i = index; i < N; i += step)
    {
        mat1[i] = mat1[i] / mat2[i];
    }
}

__global__ void moyGPU1(double* res, double* mat1, double* nb, const int line, const int column) //vecteur ligne
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int step = blockDim.x * gridDim.x;
     
    for (int i = index; i < column; i += step)
    {
        double s = 0.0;
        for (int j = 0; j < line; j++)
        {
            s += mat1[i + column *j];
        }
        res[i] = s / nb[i];
    }

}
__global__ void moyGPU2(double* res, double* mat1, double* nb, const int line, const int column) // vecteur colonne
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int step = blockDim.x * gridDim.x;
    
    for (int i = index; i < line; i += step)
    {
        double s = 0.0;
        for (int j = 0; j < column; j++)
        {
            s +=  mat1[i*column + j];
        }
        res[i] = s /nb[i];
    }
}

__global__ void projectGPU(double* mat, double* Lb, double* Ub, int N) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int step = blockDim.x * gridDim.x;
    for (int i = index; i < N; i += step)
    {
        double r = mat[i];
        double ub = Ub[i];
        double lb = Lb[i];
        r = (ub - r) * (r > ub) + (lb - r) * (r < lb) + r;
        mat[i] = r;//(Ub[i] - mat[i])* (mat[i] > Ub[i]) + (Lb[i] - mat[i]) * (mat[i] < Lb[i]) + mat[i];
    }
}

__global__ void projectGPUPos(double* mat, int N) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int step = blockDim.x * gridDim.x;
    for (int i = index; i < N; i += step)
    {
        double r = mat[i];
        mat[i] = (r > 0) * r;
    }
}

__global__ void projectGPUNeg(double* mat, int N) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int step = blockDim.x * gridDim.x;
    for (int i = index; i < N; i += step)
    {
        double r = mat[i];
        mat[i] = (r < 0) * r;
    }
}


__global__ void sumGPU(double* res, double* mat1, const int line, const int column) //vecteur colonne
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int step = blockDim.x * gridDim.x;

    for (int i = index; i < line; i += step)
    {
        double s = 0.0;
        for (int j = 0; j < column; j++)
        {
            s += mat1[i*column + j];
        }
        res[i] = s;
    }
}

__global__ void sumGPU2(double* res, double* mat1, const int line) //vecteur ligne
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if(index==0)
    {
        double s = 0.0;
        for (int j = 0; j < line; j++)
        {
            s += mat1[j];
        }
        
        *res = s ;
    }
}



__device__ int sumCommSingleWarp(volatile double* shArr) {
    int idx = threadIdx.x % warpSizeD; //the lane index in the warp
    if (idx < 16) {
        shArr[idx] += shArr[idx + 16];
        shArr[idx] += shArr[idx + 8];
        shArr[idx] += shArr[idx + 4];
        shArr[idx] += shArr[idx + 2];
        shArr[idx] += shArr[idx + 1];
    }
    return shArr[0];
}

template <unsigned int blockSize>
__global__ void sumMonoBlock(double* g_idata, double* g_odata, unsigned int n) {
    
    int idx = threadIdx.x;
    double sum = 0;
    for (int i = idx; i < n; i += blockSize)
        sum += g_idata[i];
    __shared__ double r[blockSize];
    r[idx] = sum;
    sumCommSingleWarp(&r[idx & ~(warpSizeD - 1)]);
    __syncthreads();
    if (idx < warpSizeD) { //first warp only
        r[idx] = idx * warpSizeD < blockSize ? r[idx * warpSizeD] : 0;
        sumCommSingleWarp(r);
        if (idx == 0)
            *g_odata = r[0];
    }
}


template <unsigned int blockSize>
__global__ void SumMultiBlock(double* g_idata, double* g_odata, unsigned int n) {

    __shared__ double shArr[blockSize];
    int thIdx = threadIdx.x;
    int gthIdx = thIdx + blockIdx.x * blockSize;
    const int gridSize = blockSize * gridDim.x;
    double sum = 0;
    for (int i = gthIdx; i < n; i += gridSize)
       sum += g_idata[i];
    
    shArr[thIdx] = sum;
    __syncthreads();
    for (int size = blockSize / 2; size > 0; size /= 2) { //uniform
        if (thIdx < size)
            shArr[thIdx] += shArr[thIdx + size];
        __syncthreads();
    }
    if (thIdx == 0) {
        g_odata[blockIdx.x] = shArr[0];
    }
}

template <unsigned int blockSize>
__global__ void SumMultiBlock(double* g_idata, double* g_odata, unsigned int begin, unsigned int end) {
    __shared__ float shArr[blockSize];
    int thIdx = threadIdx.x;
    int gthIdx = thIdx + blockIdx.x * blockSize;
    const int gridSize = blockSize * gridDim.x;
    double sum = 0;
    for (int i = gthIdx + begin; i < end; i += gridSize)
        sum += g_idata[i];

    shArr[thIdx] = sum;
    __syncthreads();
    for (int size = blockSize / 2; size > 0; size /= 2) { //uniform
        if (thIdx < size)
            shArr[thIdx] += shArr[thIdx + size];
        __syncthreads();
    }
    if (thIdx == 0) {
        g_odata[blockIdx.x] = shArr[0];
    }
}


template <unsigned int blockSize>
__global__ void SumEachRow(double* g_idata, double* g_odata, const int nCol) {
    __shared__ double shArr[blockSize];
    int thIdx = threadIdx.x;
    int row = blockIdx.x;
    int idBegin = thIdx + row * nCol;
    int idEnd = (row + 1) * nCol;
    int step = blockDim.x;

    double sum = 0;
    for (int i = idBegin; i < idEnd; i += step) {
        sum += g_idata[i]; 
    }
        

    shArr[thIdx] = sum;
    __syncthreads();
    for (int size = blockSize / 2; size > 0; size /= 2) { //uniform
        if (thIdx < size)
            shArr[thIdx] += shArr[thIdx + size];
        __syncthreads();
    }
    if (thIdx == 0) {
        g_odata[blockIdx.x] = shArr[0];
    }
}

template <unsigned int blockSize>
__global__ void distanceMultiBlock(double* g_idata, double* g_odata, unsigned int n) {
    int thIdx = threadIdx.x;
    int gthIdx = thIdx + blockIdx.x * blockSize;
    const int gridSize = blockSize * gridDim.x;
    double sum = 0;
    for (int i = gthIdx; i < n; i += gridSize)
        sum += (g_idata[i] * g_idata[i]);
    __shared__ double shArr[blockSize];
    shArr[thIdx] = sum;
    __syncthreads();
    for (int size = blockSize / 2; size > 0; size /= 2) { //uniform
        if (thIdx < size)
            shArr[thIdx] += shArr[thIdx + size];
        __syncthreads();
    }
    if (thIdx == 0)
        g_odata[blockIdx.x] = shArr[0];
   
}



template <unsigned int blockSize>
__global__ void distanceMultiBlock(double* g_idata, double* g_idata2, double* g_odata, unsigned int n) {
    int thIdx = threadIdx.x;
    int gthIdx = thIdx + blockIdx.x * blockSize;
    const int gridSize = blockSize * gridDim.x;
    double sum = 0;
    for (int i = gthIdx; i < n; i += gridSize)
        sum += ((g_idata[i]- g_idata2[i]) * (g_idata[i] - g_idata2[i]));
    __shared__ double shArr[blockSize];
    shArr[thIdx] = sum;
    __syncthreads();
    for (int size = blockSize / 2; size > 0; size /= 2) { //uniform
        if (thIdx < size)
            shArr[thIdx] += shArr[thIdx + size];
        __syncthreads();
    }
    if (thIdx == 0)
        g_odata[blockIdx.x] = shArr[0];

}

__device__ double warpReduceMax(volatile double* r) {
    int idx = threadIdx.x % warpSizeD; //the lane index in the warp
    if (idx < 16) {
        r[idx] = r[idx + 16] > r[idx] ? r[idx + 16] : r[idx];//r[idx + 16] > r[idx] ? r[idx + 16] : r[idx];//r[idx + 16] * (r[idx + 16] > r[idx]) + r[idx] * (r[idx] <= r[idx + 16]);
        r[idx] = r[idx + 8] > r[idx] ? r[idx + 8] : r[idx];//r[idx +  8] > r[idx] ? r[idx +  8] : r[idx];//r[idx +  8] * (r[idx +  8] > r[idx]) + r[idx] * (r[idx] <= r[idx +  8]);
        r[idx] = r[idx + 4] > r[idx] ? r[idx + 4] : r[idx];//r[idx +  4] > r[idx] ? r[idx +  4] : r[idx];//r[idx +  4] * (r[idx +  4] > r[idx]) + r[idx] * (r[idx] <= r[idx +  4]);
        r[idx] = r[idx + 2] > r[idx] ? r[idx + 2] : r[idx];//r[idx +  2] > r[idx] ? r[idx +  2] : r[idx];//r[idx +  2] * (r[idx +  2] > r[idx]) + r[idx] * (r[idx] <= r[idx +  2]);
        r[idx] = r[idx + 1] > r[idx] ? r[idx + 1] : r[idx];//r[idx +  1] > r[idx] ? r[idx +  1] : r[idx];//r[idx +  1] * (r[idx +  1] > r[idx]) + r[idx] * (r[idx] <= r[idx +  1]);
    }
    return r[0];
}

__device__ void warpReduceMax(volatile double* r, volatile int* pos){
    int idx = threadIdx.x % warpSizeD; //the lane index in the warp
    if (idx < 16) {
        pos[idx] = r[idx + 16] > r[idx] ? pos[idx + 16] : pos[idx];
        r[idx] = r[idx + 16] > r[idx] ? r[idx + 16] : r[idx];//r[idx + 16] > r[idx] ? r[idx + 16] : r[idx];//r[idx + 16] * (r[idx + 16] > r[idx]) + r[idx] * (r[idx] <= r[idx + 16]);
        
        pos[idx] = r[idx + 8] > r[idx] ? pos[idx + 8] : pos[idx];
        r[idx] = r[idx + 8] > r[idx] ? r[idx + 8] : r[idx];//r[idx +  8] > r[idx] ? r[idx +  8] : r[idx];//r[idx +  8] * (r[idx +  8] > r[idx]) + r[idx] * (r[idx] <= r[idx +  8]);
        
        pos[idx] = r[idx + 4] > r[idx] ? pos[idx + 4] : pos[idx];
        r[idx] = r[idx + 4] > r[idx] ? r[idx + 4] : r[idx];//r[idx +  4] > r[idx] ? r[idx +  4] : r[idx];//r[idx +  4] * (r[idx +  4] > r[idx]) + r[idx] * (r[idx] <= r[idx +  4]);
        
        pos[idx] = r[idx + 2] > r[idx] ? pos[idx + 2] : pos[idx];
        r[idx] = r[idx + 2] > r[idx] ? r[idx + 2] : r[idx];//r[idx +  2] > r[idx] ? r[idx +  2] : r[idx];//r[idx +  2] * (r[idx +  2] > r[idx]) + r[idx] * (r[idx] <= r[idx +  2]);
        
        pos[idx] = r[idx + 1] > r[idx] ? pos[idx + 1] : pos[idx];
        r[idx] = r[idx + 1] > r[idx] ? r[idx + 1] : r[idx];//r[idx +  1] > r[idx] ? r[idx +  1] : r[idx];//r[idx +  1] * (r[idx +  1] > r[idx]) + r[idx] * (r[idx] <= r[idx +  1]);
    }
}

template <unsigned int blockSize>
__global__ void maxMonoBlock(double* g_idata, double* g_odata, unsigned int n) {
    int idx = threadIdx.x;
    double max = 0;
    for (int i = idx; i < n; i += blockSize) {
        double s = g_idata[i];
        max = s > max ? s : max;// s>max ? s:max;//s * (s > max) + max * (max <= s);
    }
    __shared__ double r[blockSize];
    r[idx] = max;
    warpReduceMax(&r[idx & ~(warpSizeD - 1)]);
    __syncthreads();
    if (idx < warpSizeD) { //first warp only
        r[idx] = idx * warpSizeD < blockSize ? r[idx * warpSizeD] : 0;
        warpReduceMax(r);
        if (idx == 0)
            *g_odata = r[0];
    }
}
template <unsigned int blockSize>
__global__ void maxMultiBlock(double* g_idata, double* g_odata, unsigned int n) {
    int thIdx = threadIdx.x;
    int gthIdx = thIdx + blockIdx.x * blockSize;
    const int gridSize = blockSize * gridDim.x;
    double max = 0;
    for (int i = gthIdx; i < n; i += gridSize) {
        double s = (g_idata[i] * g_idata[i]);
        max = s > max ? s : max;//s > max ? s : max; //s * (s > max) + max * (max <= s);
    }
    __shared__ double shArr[blockSize];
    shArr[thIdx] = max;
    __syncthreads();
    for (int size = blockSize / 2; size > 0; size /= 2) { //uniform
        if (thIdx < size)
            shArr[thIdx] = shArr[thIdx + size] > shArr[thIdx] ? shArr[thIdx + size] : shArr[thIdx]; //shArr[thIdx + size] > shArr[thIdx] ? shArr[thIdx + size] : shArr[thIdx];//shArr[thIdx + size] * (shArr[thIdx + size] > shArr[thIdx]) + shArr[thIdx] * (shArr[thIdx] <= shArr[thIdx + size]); 
        __syncthreads();
    }
    if (thIdx == 0)
        g_odata[blockIdx.x] = shArr[0];
}

template <unsigned int blockSize>
__global__ void maxMultiBlock(double* g_idata, double* g_odata, unsigned int n, int* pos) {
    int thIdx = threadIdx.x;
    int gthIdx = thIdx + blockIdx.x * blockSize;
    const int gridSize = blockSize * gridDim.x;
    double max = 0;
    int indice = 0;
    for (int i = gthIdx; i < n; i += gridSize) {
        double s = (g_idata[i] * g_idata[i]);
        indice = s > max ? i : indice;
        max = s > max ? s : max;//s > max ? s : max; //s * (s > max) + max * (max <= s);
    }
    __shared__ double shArr[blockSize];
    __shared__ double shPos[blockSize];
    shArr[thIdx] = max;
    shPos[thIdx] = indice;
    __syncthreads();
    for (int size = blockSize / 2; size > 0; size /= 2) { //can unroll the for loop
        if (thIdx < size) {
            shPos[thIdx] = shArr[thIdx + size] > shArr[thIdx] ? shPos[thIdx + size] : shPos[thIdx];
            shArr[thIdx] = shArr[thIdx + size] > shArr[thIdx] ? shArr[thIdx + size] : shArr[thIdx]; 
            //shArr[thIdx + size] > shArr[thIdx] ? shArr[thIdx + size] : shArr[thIdx];//shArr[thIdx + size] * (shArr[thIdx + size] > shArr[thIdx]) + shArr[thIdx] * (shArr[thIdx] <= shArr[thIdx + size]); 
        }
           
        __syncthreads();
    }
    if (thIdx == 0) {
        g_odata[blockIdx.x] = shArr[0];
        pos[blockIdx.x] = shPos[0];

    }
}
       

template <unsigned int blockSize>
__global__ void maxMonoBlock(double* g_idata, double* g_odata, unsigned int n, int* pos) {
    int idx = threadIdx.x;
    double max = 0;
    int indice = 0;
    for (int i = idx; i < n; i += blockSize) {
        double s = g_idata[i];
        indice = s > max ? pos[i] : indice;
        max = s > max ? s : max;// s>max ? s:max;//s * (s > max) + max * (max <= s);
    }
    __shared__ double r[blockSize];
    __shared__ int shPos[blockSize];
    r[idx] = max;
    shPos[idx] = indice;
    warpReduceMax(&r[idx & ~(warpSizeD - 1)], &shPos[idx & ~(warpSizeD - 1)]);
    __syncthreads();
    if (idx < warpSizeD) { //first warp only
        r[idx] = idx * warpSizeD < blockSize ? r[idx * warpSizeD] : 0;
        warpReduceMax(r, shPos);
        if (idx == 0) {
            *g_odata = r[0];
            *pos = shPos[0];
        }
            
        
    }
}


template <unsigned int blockSize>
__global__ void maxMultiBlock(double* g_idata, double* g_idata2, double* g_odata, unsigned int n) {
    int thIdx = threadIdx.x;
    int gthIdx = thIdx + blockIdx.x * blockSize;
    const int gridSize = blockSize * gridDim.x;
    double max = 0;
    for (int i = gthIdx; i < n; i += gridSize) {
        double s = (g_idata[i] - g_idata2[i]);
        s = s*s;
        max = s > max ? s : max;//s > max ? s : max; //s * (s > max) + max * (max <= s);
    }
    __shared__ double shArr[blockSize];
    shArr[thIdx] = max;
    __syncthreads();
    for (int size = blockSize / 2; size > 0; size /= 2) { //uniform
        if (thIdx < size) {
            shArr[thIdx] = shArr[thIdx + size] > shArr[thIdx] ? shArr[thIdx + size] : shArr[thIdx]; //shArr[thIdx + size] > shArr[thIdx] ? shArr[thIdx + size] : shArr[thIdx];//shArr[thIdx + size] * (shArr[thIdx + size] > shArr[thIdx]) + shArr[thIdx] * (shArr[thIdx] <= shArr[thIdx + size]); 
        }
            
        __syncthreads();
    }
    if (thIdx == 0) {
        g_odata[blockIdx.x] = shArr[0];
    }
}


__global__ void normalisationGJ(double* mat, const int row, const int nCol, const double factor) 
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int step = blockDim.x * gridDim.x;
    for (int i = index; i < nCol; i += step)
    {
        mat[i + row * nCol] = mat[i + row * nCol] / factor;
    }


}

__global__ void swapLineGJ(double* mat, const int row1, const int row2, const int nCol) 
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int step = blockDim.x * gridDim.x;
    for (int i = index; i < nCol; i += step)
    {
        double temp = mat[i + row1 * nCol];
        double temp2 = mat[i + row2 * nCol]; 
        mat[i + row1 * nCol] = temp2; // or mat[i + row1 * nCol] = mat[i + row2 * nCol];
        mat[i + row2 * nCol] = temp;
    }
}

__global__ void eliminationGJ(double* mat, double* matAug, const int r, const int nRow, const int nCol) {

    // un bloc = une ligne, 
    int index = threadIdx.x;
    int row = blockIdx.x;
    int step = blockDim.x;
    __shared__ double shFactor;
    if (row != r) { // le bloc r ne fait rien... bah...
        if (index == 0) {
            shFactor = mat[row * nCol + r];
        }
        __syncthreads();
        for (int j = index; j < nCol; j+=step) {
            double value1 = mat[r * nCol + j];
            double oldvalue1 = mat[row * nCol + j];
            double oldvalue2 = matAug[row * nCol + j];
            double value2 = matAug[r * nCol + j];

            mat[row * nCol + j] = oldvalue1 - shFactor * value1;
            //matAug[row * nCol + j] -=  shFactor * value2;
            matAug[row * nCol + j] = oldvalue2 - shFactor * value2;
        }
    }
    

    /*int indexX = blockIdx.x * blockDim.x + threadIdx.x;
    int indexY = blockIdx.y * blockDim.y + threadIdx.y;
    int stepX = blockDim.x * gridDim.x;
    int stepY = blockDim.y * gridDim.y;


    for (int i = indexY; i < nRow; i += stepY)
    {
        if (i != r) {
            double factor = mat[i * nCol + r]; // ne doit pas changer tant que la ligne n'est pas fini
            for (int j = indexX; j < nCol; j += stepX)
            {
                if (j != r) {
                    double value1 = mat[r * nCol + j];
                    double value2 = matAug[r * nCol + j];

                    mat[i * nCol + j] = mat[i * nCol + j] - factor * value1;
                    matAug[i * nCol + j] = matAug[i * nCol + j] - factor * value2;
                }
            }
        }
    }
    __syncthreads();
    for (int i = indexY; i < nRow; i += stepY)
    {
        double factor = mat[i * nCol + r]; // ne doit pas changer tant que la ligne n'est pas fini
        if (i != r) {
            if (indexX == 1) {
                mat[i * nCol + r] = 0;
                matAug[i * nCol + r] = matAug[i * nCol + r] - factor * matAug[r * nCol + r];
            }
        }
    }*/
}



__global__ void initPermMatr(double* P, const int N) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int step = blockDim.x * gridDim.x;
    for (int i = index; i < (N + 1); i += step)
    {
        P[i] = i * (i < N);
    }
}


__global__ void updatePermMatr(double* P, const int line1, const int line2, const int N) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int step = blockDim.x * gridDim.x;
    if (index == 0) {
        int inter = P[line1];
        P[line1] = P[line2];
        P[line2] = inter;
        P[N] = P[N] + 1;
    }
}


__global__ void updateLUPFactorization(double* A, const int col, const int N) {
    // un bloc par ligne i ?
    int index = threadIdx.x;
    int i = blockIdx.x;
    int step = blockDim.x;

    __shared__ double Aicol;

    if (i > col) { // les blocs trop petits ne font rien, en s'en fout ?
        if (index == 0) {
            Aicol = A[i * N + col] / A[col * N + col];
            A[i * N + col] = Aicol;
        }
        __syncthreads();
        for (int k = index + col + 1; k < N; k += step) {
            A[i * N + k] = A[i * N + k] - Aicol * A[col * N + k];
        }
    }
}





__global__ void setPermute(double* y, double* b, double* P, const int N) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int step = blockDim.x * gridDim.x;

    for (int i = index; i < N; i += step)
    {
        int indice = P[i];
        y[i] = b[indice]; // c'est absolument moche...
    }


}
__global__ void solveLowSys(double* A, double* y, const int iter, const int N) {
    int index = threadIdx.x;
    int step = blockDim.x;
    __shared__ double yiter;

    if (index == 0) {
        yiter = y[iter];

    }
    __syncthreads();
    for (int i = index + iter + 1; i < N; i += step)
    {
        y[i] = y[i] - y[iter] * A[i * N + iter]; // moche ne faudrait-il pas stocker A^T ?

    }


}

__global__ void solveUpSys(double* A, double* y, const int iter, const int N) {
    int index = threadIdx.x;
    int step = blockDim.x;
    __shared__ double yiter;

    if (index == 0) {
        yiter = y[iter] / A[iter * N + iter];
        y[iter] = yiter;
    }
    __syncthreads();

    for (int i = index; i < iter; i += step)
    {
        y[i] = y[i] - yiter * A[i * N + iter]; // moche ne faudrait-il pas stocker A^T ?

    }


}

__global__ void solveSysGPU(double* A, double* y, const int N) {


    int index = threadIdx.x;
    int step = blockDim.x;
    extern __shared__ double ytemp[];


    for (int n = index; n < N; n += step)
    {
        ytemp[n] = y[n];
    }
    __syncthreads();
    for (int iter = 0; iter < N; iter++) {
        for (int i = index + iter + 1; i < N; i += step)
        {
            ytemp[i] = ytemp[i] - ytemp[iter] * A[i * N + iter]; // moche ne faudrait-il pas stocker A^T ?

        }
        __syncthreads();
    }
    for (int iter = N - 1; iter >= 0; iter--) {
        if (index == 0) {
            ytemp[iter] = ytemp[iter] / A[iter * N + iter];
            y[iter] = ytemp[iter];
        }
        __syncthreads();

        for (int i = index; i < iter; i += step)
        {
            ytemp[i] = ytemp[i] - ytemp[iter] * A[i * N + iter]; // moche ne faudrait-il pas stocker A^T ?

        }
        __syncthreads();
    }
}