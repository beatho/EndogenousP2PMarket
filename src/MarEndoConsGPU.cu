#include "hip/hip_runtime.h"
#include "../head/MarEndoConsGPU.cuh"




MarEndoConsGPU::MarEndoConsGPU() : MethodP2PGPU()
{
#if DEBUG_CONSTRUCTOR
	std::cout << " MarEndoConsGPU Constructor" << std::endl;
#endif // DEBUG_CONSTRUCTOR
	_name = NAME;
	timePerBlock = MatrixCPU(1, 9, 0); // Fb0, Fb1 , Fb2, Fb3, Fb5, Fb6 Fb0'
	// si les sous ensemble ne sont pas accessible, tout est dans le premier.
	occurencePerBlock = MatrixCPU(1, 9, 0); //nb de fois utilis� pendant la simu
}


MarEndoConsGPU::MarEndoConsGPU(float rho) : MethodP2PGPU()
{
#if DEBUG_CONSTRUCTOR
	std::cout << "default MarEndoConsGPU Constructor" << std::endl;
#endif // DEBUG_CONSTRUCTOR
	_name = NAME;
	_rho = rho;
	timePerBlock = MatrixCPU(1, 9, 0); // Fb0, Fb1 , Fb2, Fb3, Fb5, Fb6 Fb0'
	// si les sous ensemble ne sont pas accessible, tout est dans le premier.
	occurencePerBlock = MatrixCPU(1, 9, 0); //nb de fois utilis� pendant la simu
}

MarEndoConsGPU::~MarEndoConsGPU()
{
	DELETEB(OPF);
	//DELETEB(OPFCPU);
}


void MarEndoConsGPU::setParam(float rho)
{
	_rho = rho;
}

void MarEndoConsGPU::setTau(float tau)
{
	if (tau < 1) {
		throw std::invalid_argument("tau must be greater than 1");
	}
	_tau = tau;
}



void MarEndoConsGPU::solve(Simparam* result, const Simparam& sim, const StudyCase& cas)
{
#ifdef DEBUG_SOLVE
	cas.display();
	sim.display(1);
#endif // DEBUG_SOLVE
	
	clock_t tall = clock();
#ifdef INSTRUMENTATION
	std::chrono::high_resolution_clock::time_point t1;
	std::chrono::high_resolution_clock::time_point t2;
#endif // INSTRUMENTATION


	// FB 0
	if (_id == 0) {
#ifdef INSTRUMENTATION
		t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION
		init(sim, cas);
#ifdef INSTRUMENTATION
		hipDeviceSynchronize();
		t2 = std::chrono::high_resolution_clock::now();
		timePerBlock.set(0, 0, (float) std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
		occurencePerBlock.set(0, 0, 1);
#endif // INSTRUMENTATION
	}
	
	_rhog = sim.getRho();
	_at1 = _rhog;
	_iterG = sim.getIterG();
	int iterL = sim.getIterL();
	int stepL = sim.getStepL();

	
	float epsL = sim.getEpsL() / 20;
	float epsG = sim.getEpsG();
	

	float fc = 0;

	
	_resG = 2 * epsG;
	float epsL2 = epsL * epsL;

	_iterGlobal = 0;
	while ((_iterGlobal < _iterG) && (_resG > epsG)  || (_iterGlobal <=_stepG)) { // || (_iterGlobal <= _stepG)
		//std::cout << "*";
		/*std::cout << "---------------------------------" << std::endl;
		std::cout << " Pn " << std::endl;
		Pn.display(true);
		std::cout << " Pso " << std::endl;
		PSO.display(true);
		std::cout << " Bp3 " << std::endl;
		Bp3.display(true);*/

		/*std::cout << " Tlocal " << std::endl;
		Tlocal.display(true);
		std::cout << " Bt1 " << std::endl;
		Bt1.display(true);
		std::cout << " matlb " << std::endl;
		matLb.display(true);
		std::cout << " matUb " << std::endl;
		matUb.display(true); */
		

#ifdef INSTRUMENTATION
		hipDeviceSynchronize();
		t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION
		//CHECK_LAST_CUDA_ERROR();
		updateLocalProbGPU(epsL2, iterL);
		//CHECK_LAST_CUDA_ERROR();
#ifdef INSTRUMENTATION
		hipDeviceSynchronize();
		t2 = std::chrono::high_resolution_clock::now();
		timePerBlock.increment(0, 1, (float) std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());

#endif // INSTRUMENTATION
		//std::cout << _iterGlobal << " " << iterLocal << " " << resL << " " << resF.get(0, _iterGlobal / _stepG) << " " << resF.get(1, _iterGlobal / _stepG) << std::endl;

		tradeLin.swap(&Tlocal);
		updateGlobalProb();
		//CHECK_LAST_CUDA_ERROR();
		// FB 4
		if (!(_iterGlobal % _stepG)) {
#ifdef INSTRUMENTATION
			t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION
			_resG = updateResBis( _iterGlobal / _stepG );
			//std::cout << _iterGlobal << " " << iterLocal << " " << resL << " " << resF.get(0, _iterGlobal / _stepG)
				//<< " " << resF.get(1, _iterGlobal / _stepG) << " " << resF.get(2, _iterGlobal / _stepG) << std::endl;
#ifdef INSTRUMENTATION
			hipDeviceSynchronize();
			t2 = std::chrono::high_resolution_clock::now();
			timePerBlock.increment(0, 6, (float) std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
#endif // INSTRUMENTATION		
		}
		//CHECK_LAST_CUDA_ERROR();

		_iterGlobal++;
	}
	
	//std::cout << _iterGlobal << " " << iterLocal  << " " << resF.get(0, (_iterGlobal - 1) / _stepG) << " " << resF.get(1, (_iterGlobal - 1) / _stepG) << " " << resF.get(2, (_iterGlobal - 1) / _stepG) << std::endl;
#ifdef INSTRUMENTATION
	occurencePerBlock.increment(0, 1, _iterGlobal);
	occurencePerBlock.increment(0, 3, _iterGlobal);
	occurencePerBlock.increment(0, 4, _iterGlobal);
	occurencePerBlock.increment(0, 5, _iterGlobal);
	occurencePerBlock.increment(0, 6, _iterGlobal / _stepG);

	t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION
	
	fc = calcFc();
	
	MatrixCPU tradeLinCPU;
	tradeLin.toMatCPU(tradeLinCPU);
	MatrixCPU LAMBDALinCPU;
	LAMBDALin.toMatCPU(LAMBDALinCPU);
	MatrixCPU PnCPU;
	Pn.toMatCPU(PnCPU);
	MatrixCPU MUCPU;
	MU.toMatCPU(MUCPU);


	int indice = 0;
	for (int idAgent = 0; idAgent < _nAgentTrue; idAgent++) {
		MatrixGPU omega(cas.getVoisin(idAgent));
		int Nvoisinmax = nVoisinCPU.get(idAgent, 0);
		for (int voisin = 0; voisin < Nvoisinmax; voisin++) {
			int idVoisin = omega.get(voisin, 0);
			trade.set(idAgent, idVoisin, tradeLinCPU.get(indice, 0));
			LAMBDA.set(idAgent, idVoisin, LAMBDALinCPU.get(indice, 0));
			indice = indice + 1;
		}
	}
	for (int idAgent = _nAgentTrue; idAgent < _nAgent; idAgent++) {
		for (int idVoisin = 0; idVoisin < _nAgentTrue; idVoisin++) {
			if (idVoisin != (idAgent - _nAgentTrue)) {
				trade.set(idAgent, idVoisin, tradeLinCPU.get(indice, 0));
				LAMBDA.set(idAgent, idVoisin, LAMBDALinCPU.get(indice, 0));
				indice = indice + 1;
			}

		}
	}
	//trade.display();
	/*std::cout << "PSO, Pn" << std::endl;
	PSO.display();
	Pn.display();*/

	// FB 5
	
	MatrixCPU Pb(OPF->getPb());
	MatrixCPU Phi(OPF->getPhi());
	MatrixCPU E(OPF->getE());
	
	result->setE(&E);
	result->setPhi(&Phi);
	result->setPb(&Pb);

	result->setResF(&resF);
	
	result->setLAMBDA(&LAMBDA);
	
	result->setTrade(&trade);
	result->setIter(_iterGlobal);
	
	result->setPn(&PnCPU);
	result->setFc(fc);
	result->setMU(&MUCPU);

#ifdef INSTRUMENTATION
	hipDeviceSynchronize();
	t2 = std::chrono::high_resolution_clock::now();
	timePerBlock.increment(0, 7, (float) std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
	occurencePerBlock.increment(0, 7, 1);
	result->setTimeBloc(&timePerBlock, &occurencePerBlock);
#endif // INSTRUMENTATION
	timeMarketEndo = clock() - tall;
	result->setTime((float) timeMarketEndo / CLOCKS_PER_SEC);


	//std::cout << "****" << std::endl;
	//OPF->display();
	//display();
}

void MarEndoConsGPU::updateP0(const StudyCase& cas)
{
	_id = _id + 1;
#ifdef INSTRUMENTATION
	std::chrono::high_resolution_clock::time_point t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION
	
	matLb.transferCPU();
	matUb.transferCPU();

	Pmin = cas.getPmin();
	Pmax = cas.getPmax();


	MatrixCPU Lb(cas.getLb());
	MatrixCPU Ub(cas.getUb());

	b = cas.getb();
	Cp = b;
	int indice = 0;

	for (int idAgent = 0; idAgent < _nAgent; idAgent++) {
		int Nvoisinmax = nVoisinCPU.get(idAgent, 0);
		for (int voisin = 0; voisin < Nvoisinmax; voisin++) {
			matLb.set(indice, 0, Lb.get(idAgent, 0));
			matUb.set(indice, 0, Ub.get(idAgent, 0));
			indice = indice + 1;
		}
	}
	matLb.transferGPU();
	matUb.transferGPU();

	float Ploss = Pn.get(0, 0, false);
	float Qloss = Pn.get(_nAgentTrue, 0, false);
	// pour essayer que cela marche
	Pn.add(&Pmin, &Pmax);
	Pn.divide(2);

	// unleash powe
	Pmin.set(0, 0, -POWERLIMIT, true);
	Pmax.set(_nAgentTrue, 0, POWERLIMIT, true);
	Pmin.set(_nAgentTrue, 0, -POWERLIMIT, true);

	Pn.set(0, 0, Ploss, true);
	Pn.set(_nAgentTrue, 0, Qloss, true);

	Pmin.divideT(&nVoisin);
	Pmax.divideT(&nVoisin);
	Cp.multiplyT(&nVoisin);
#ifdef INSTRUMENTATION
	hipDeviceSynchronize();
	std::chrono::high_resolution_clock::time_point t2 = std::chrono::high_resolution_clock::now();
	timePerBlock.increment(0, 8, (float) std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
	occurencePerBlock.increment(0, 8, 1);
#endif // INSTRUMENTATION


}

void MarEndoConsGPU::init(const Simparam& sim, const StudyCase& cas)
{
	DELETEB(OPF);
	DELETEB(OPFCPU);
	


	if (CoresMatLin.getPos()) { // une copie en trop mais pour l'instant c'est ok...
		CoresMatLin.transferCPU();
		CoresLinAgent.transferCPU();
		CoresAgentLin.transferCPU();
		CoresLinVoisin.transferCPU();
		CoresLinTrans.transferCPU();

		Tlocal_pre.transferCPU();
		tradeLin.transferCPU();
		LAMBDALin.transferCPU();

		matLb.transferCPU();
		matUb.transferCPU();
		Ct.transferCPU();

	}
	//std::cout << "init " << std::endl;
	//hipDeviceSynchronize();
	//CHECK_LAST_CUDA_ERROR();
	// intitilisation des matrixs et variables 
	
	if (!cas.isAC()) {
		throw std::invalid_argument("Wrong studyCase must be AC");
	}
	if (!cas.isRadial()) {
		throw std::invalid_argument("Wrong studyCase must be radial, dont have OPF on non-radial cases");
	}
	////CHECK_LAST_CUDA_ERROR();
	_rhog = sim.getRho();
	_rhoSO = _rhog;
	//_rhoSO = sim.getRho1();
	_iterG = sim.getIterG();
	_stepG = sim.getStepG();
	float epsG = sim.getEpsG();
	float epsGC = sim.getEpsGC();
	_epsLim = sim.getEpsIntern();
	_stepL = sim.getStepL();
	_ratioEps = epsG / epsGC;
	_nAgentTrue = sim.getNAgent();
	if (_nAgentTrue != cas.getNagent()) {
		throw std::invalid_argument("nAgent different on Simparam and study case");
	}
	_nAgent = 2 * _nAgentTrue;

	paramOPF = sim;
	paramOPF.setItG(sim.getIterIntern());
	paramOPF.setEpsG(sim.getEpsIntern());
	_stepIntern = sim.getStepIntern();


	_rhol = _rho; //*nAgent
	
	//std::cout << "rho " << _rho << std::endl;
	if (_rho == 0) {
		_rhol = _rhog;
	}
	
	CHECK_LAST_CUDA_ERROR();
	nVoisinCPU = cas.getNvoi();
	nVoisin = MatrixGPU(nVoisinCPU, 1);
	nVoisin.preallocateReduction();
	int nVoisinMax = nVoisin.max2();
	if (_blockSize * NMAXPEERPERTRHREAD < nVoisinMax) {
		std::cout << _blockSize << " " << NMAXPEERPERTRHREAD << " " << nVoisinMax << std::endl;
		throw std::invalid_argument("For this Method, an agent must not have more than 5120 peers");
	}
	//std::cout << "Trades" << std::endl;
	_nTrade = nVoisin.sum();
	_nTradeP = nVoisin.sum(0, _nAgentTrue);
	
	_nTradeQ = _nTrade - _nTradeP;
	if (_nTradeQ != (_nAgentTrue * (_nAgentTrue - 1))) {
		std::cout << "err MarEndoGPU : " << _nAgent << " " << _nAgentTrue << " " << _nTrade << " " << _nTradeP << " " << _nTradeQ << std::endl;
		throw std::invalid_argument("Agent must be fully conected for the Q echanges, WIP");
	}
	_numBlocksN = ceil((_nAgent + _blockSize - 1) / _blockSize);
	_numBlocksM = ceil((_nTrade + _blockSize - 1) / _blockSize);

	/*hipDeviceSynchronize();
	//CHECK_LAST_CUDA_ERROR();*/
	//std::cout << "Market" << std::endl;
	std::chrono::high_resolution_clock::time_point t1 = std::chrono::high_resolution_clock::now();
	
	MatrixCPU PnCPU(2 * _nAgent, 1);
	if (initWithMarketClear) {
		ADMMMarketGPU market;
		Simparam res(sim);
		market.solve(&res, sim, cas);
		//res.display();
		LAMBDA = res.getLambda();
		trade = res.getTrade();
		PnCPU = res.getPn();
	}
	else {
		LAMBDA = sim.getLambda();
		trade = sim.getTrade();
		PnCPU = sim.getPn();
	}
	std::chrono::high_resolution_clock::time_point t2 = std::chrono::high_resolution_clock::now();
	
	//std::cout << "time : " <<  (float)std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count() / 1000000000 << std::endl;
	//PnCPU.display();
	Pn = MatrixGPU(PnCPU, 1);
	//Pn.display(true);
	paramOPF.setPn(&PnCPU);
	
	_at1 = _rhog; 
	_at2 = _rhol;

	resF = MatrixCPU(3, (_iterG / _stepG) + 1);
	
	/*hipDeviceSynchronize();
	//CHECK_LAST_CUDA_ERROR();*/

	MatrixGPU BETA(cas.getBeta());
	
	MatrixGPU Ub(cas.getUb());
	MatrixGPU Lb(cas.getLb());
	
	
	//std::cout << "mise sous forme lin�aire" << std::endl;
	
	CoresMatLin = MatrixGPU(_nAgent, _nAgentTrue, -1);
	CoresAgentLin = MatrixGPU(_nAgent + 1, 1);
	CoresLinAgent = MatrixGPU(_nTrade, 1);
	CoresLinVoisin = MatrixGPU(_nTrade, 1);
	CoresLinTrans = MatrixGPU(_nTrade, 1);

	Tlocal_pre = MatrixGPU(_nTrade, 1);
	tradeLin = MatrixGPU(_nTrade, 1);
	LAMBDALin = MatrixGPU(_nTrade, 1);

	matLb = MatrixGPU(_nTrade, 1);
	matUb = MatrixGPU(_nTrade, 1);
	Ct = MatrixGPU(_nTrade, 1);
	//Bt2 = MatrixGPU(_nTrade, 1);
	/*hipDeviceSynchronize();
	//CHECK_LAST_CUDA_ERROR();*/
	int indice = 0;

	for (int idAgent = 0; idAgent < _nAgentTrue; idAgent++) { // P
		MatrixGPU omega(cas.getVoisin(idAgent));
		int Nvoisinmax = nVoisinCPU.get(idAgent, 0);
		for (int voisin = 0; voisin < Nvoisinmax; voisin++) {
			int idVoisin = omega.get(voisin, 0);
			matLb.set(indice, 0, Lb.get(idAgent, 0));
			matUb.set(indice, 0, Ub.get(idAgent, 0));
			Ct.set(indice, 0, BETA.get(idAgent, idVoisin));
			tradeLin.set(indice, 0, trade.get(idAgent, idVoisin));
			Tlocal_pre.set(indice, 0, trade.get(idAgent, idVoisin));
			LAMBDALin.set(indice, 0, LAMBDA.get(idAgent, idVoisin));
			CoresLinAgent.set(indice, 0, idAgent);
			CoresLinVoisin.set(indice, 0, idVoisin);
			CoresMatLin.set(idAgent, idVoisin, indice);
			indice = indice + 1;
		}
		CoresAgentLin.set(idAgent + 1, 0, indice);
	}
	//std::cout << " Q " << std::endl;
	for (int idAgent = _nAgentTrue; idAgent < _nAgent; idAgent++) { // Q
		for (int idVoisin = 0; idVoisin < _nAgentTrue; idVoisin++) {
			if (idVoisin != (idAgent - _nAgentTrue)) {
				if(Lb.getNCol()==1){
					matLb.set(indice, 0, Lb.get(idAgent, 0));
					matUb.set(indice, 0, Ub.get(idAgent, 0));
				} else {
					matLb.set(indice, 0, Lb.get(idAgent, idVoisin));
					matUb.set(indice, 0, Ub.get(idAgent, idVoisin));
				}
				tradeLin.set(indice, 0, trade.get(idAgent, idVoisin));
				Tlocal_pre.set(indice, 0, trade.get(idAgent, idVoisin));
				LAMBDALin.set(indice, 0, LAMBDA.get(idAgent, idVoisin));
				CoresLinAgent.set(indice, 0, idAgent);
				CoresLinVoisin.set(indice, 0, idVoisin + _nAgentTrue);
				CoresMatLin.set(idAgent, idVoisin, indice);
				indice = indice + 1;
			}
		}
		
		CoresAgentLin.set(idAgent + 1, 0, indice);
	}
	for (int lin = 0; lin < _nTrade; lin++) {
		int i = CoresLinAgent.get(lin, 0);
		int j = CoresLinVoisin.get(lin, 0);
		if (lin >= _nTradeP) {
			i -= _nAgentTrue;
		}

		int k = CoresMatLin.get(j, i);
		CoresLinTrans.set(lin, 0, k);
	}
	// transfert des mises lineaire
	matUb.transferGPU();
	matLb.transferGPU();
	Ct.transferGPU();

	Tlocal_pre.transferGPU();
	tradeLin.transferGPU();
	LAMBDALin.transferGPU();

	CoresAgentLin.transferGPU();
	CoresLinAgent.transferGPU();
	CoresLinVoisin.transferGPU();
	CoresMatLin.transferGPU();
	CoresLinTrans.transferGPU();
	CHECK_LAST_CUDA_ERROR();
	
	//CHECK_LAST_CUDA_ERROR();
	//std::cout << "donnees sur CPU pour le grid" << std::endl;

	Ap3 = nVoisin;
	Ap3.multiplyT(&nVoisin);
	Ap3.multiply(_rhoSO);
	PSO = MatrixGPU(_nAgent, 1, 0, 1);
	PSO.preallocateReduction();
	etaSO = MatrixGPU(_nAgent, 1, 0, 1);
	Bp3 = MatrixGPU(_nAgent, 1, 0, 1);
	_radial = cas.isRadial();

	 
	if (OPFonCPU) {
		OPFCPU = new OPFADMMCons;
		OPFCPU->initConsensus(paramOPF, cas, _rhoSO);
	}
	else {
		OPF = new OPFADMMConsGPU;
		OPF->initConsensus(paramOPF, cas, _rhoSO);
	}
	CHECK_LAST_CUDA_ERROR();

	
	 

	
	//CHECK_LAST_CUDA_ERROR();
	//std::cout << "autres donn�e sur CPU" << std::endl;
	tempNN = MatrixGPU(_nTrade, 1, 0, 1);
	tempNN.preallocateReduction();
	tempN1 = MatrixGPU(_nAgent, 1, 0, 1); // plut�t que de re-allouer de la m�moire � chaque utilisation
	

	Tlocal = MatrixGPU(_nTrade, 1, 0, 1);
	Tlocal.preallocateReduction();
	P = Pn; // moyenne des trades
	P.divideT(&nVoisin);

	a = MatrixGPU(cas.geta(), 1);
	b = MatrixGPU(cas.getb(), 1);

	// on enn veut pas que l'agent des pertes consomme plus que n�cessaire !!!
	//a.set(0, 0, 1);
	//a.set(_nAgentTrue, 0, 1);

	//CHECK_LAST_CUDA_ERROR();
	Ap2 = a;
	Ap1 = nVoisin;
	Ap123 = Ap3;

	Bt1 = MatrixGPU(_nTrade, 1, 0, 1);
	Cp = b;
	
	Bp1 = MatrixGPU(_nAgent, 1, 0, 1);

	Pmin = MatrixGPU(cas.getPmin(), 1);
	
	Pmin.set(0, 0, -100000, true); // unleash power !!!
	Pmin.set(_nAgentTrue, 0, -100000, true); // unleash power !!!	
	Pmax = MatrixGPU(cas.getPmax(), 1);
	Pmax.set(_nAgentTrue, 0, 100000, true); // unleash power !!!
	
	MU = MatrixGPU(sim.getMU(), 1); // facteur reduit i.e lambda_l/_rho
	Tmoy = P;

	
	Pmin.divideT(&nVoisin);
	Pmax.divideT(&nVoisin);
	Ap1.multiply(_rhol);
	Cp.multiplyT(&nVoisin);
	Tmoy.divideT(&nVoisin);
	
	Ap2.multiplyT(&nVoisin);
	Ap2.multiplyT(&nVoisin);
	Ap123.add(&Ap1);
	Ap123.add(&Ap2);
	//CHECK_LAST_CUDA_ERROR();
	/*std::cout << _at1 << " " << _at2 << std::endl;

	Ct.display(true);
	Ap1.display(true);
	Ap2.display(true);
	Ap3.display(true);
	Ap123.display(true);
	Cp.display(true);

	Pmin.display(true);
	Pmax.display(true);
	matLb.display(true);
	matUb.display(true);*/

	/*PSO.display();
	Pn.display();*/
	//std::cout << "******" << std::endl;
	//std::cout << "updateGlobal" << std::endl;
	CHECK_LAST_CUDA_ERROR();
	updateGlobalProb();
	CHECK_LAST_CUDA_ERROR();
	
	//std::cout << "fin init temps : " << (float)(clock() - t) / CLOCKS_PER_SEC << std::endl;
	//std::cout << "fin init " << std::endl;
}

void MarEndoConsGPU::updateGlobalProb() {

	// FB 3a
#ifdef INSTRUMENTATION
	std::chrono::high_resolution_clock::time_point t1 = std::chrono::high_resolution_clock::now();
	std::chrono::high_resolution_clock::time_point t2;
#endif // INSTRUMENTATION

	
	float eps = MYMIN(_resG * _delta, _epsLim);
	
	
	//std::cout << "SolveOPF" << std::endl;
	if (_iterGlobal % _stepIntern == 0) {
		if (OPFonCPU) {
			PSO.toMatCPU(PSOCPU);
			OPFCPU->solveConsensus(eps, &PSOCPU);
			PSO = PSOCPU;
		}
		else {
			OPF->solveConsensus(eps, &PSO);
		}
		CHECK_LAST_CUDA_ERROR();
	



#ifdef INSTRUMENTATION
	hipDeviceSynchronize();
	t2 = std::chrono::high_resolution_clock::now();
	timePerBlock.increment(0, 4, (float) std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
	// FB 3b
	t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION
	//std::cout << "update OPF" << std::endl;

	updatePn();
	CHECK_LAST_CUDA_ERROR();
	if (OPFonCPU) {
		Pn.toMatCPU(PnCPU);
		OPFCPU->updateConsensus(&PnCPU);
	}
	else {
		OPF->updateConsensus(&Pn);
	}
	CHECK_LAST_CUDA_ERROR();
	//MatrixCPU PnCPU;
	

	
	//
	//Pn.display();
	//PSO.display();
	
	//
	
	
	

#ifdef INSTRUMENTATION
	hipDeviceSynchronize();
	t2 = std::chrono::high_resolution_clock::now();
	timePerBlock.increment(0, 3, (float) std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
	
	t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION
	//Agent des pertes
	//std::cout << "update Market" << std::endl;
	/*Pmin.set(0, 0, Ploss / nVoisin.get(0,0));
	Pmax.set(0, 0, Ploss / nVoisin.get(0, 0));
	
	Pmin.set(_nAgentTrue, 0, Qloss / nVoisin.get(_nAgentTrue, 0));
	Pmax.set(_nAgentTrue, 0, Qloss / nVoisin.get(_nAgentTrue, 0));*/
	

	float Qloss = PSO.get(_nAgentTrue, 0, false);
	CHECK_LAST_CUDA_ERROR();
	if (Qloss > 0) {
		matUb.setBloc(_nTradeP, _nTradeP + _nAgentTrue - 1, 0, 1, Qloss);
		matLb.setBloc(_nTradeP, _nTradeP + _nAgentTrue - 1, 0, 1, 0.0);
	}
	else {
		matLb.setBloc(_nTradeP, _nTradeP + _nAgentTrue - 1, 0, 1, Qloss);
		matUb.setBloc(_nTradeP, _nTradeP + _nAgentTrue - 1, 0, 1, 0.0);
	}
	/**/
	CHECK_LAST_CUDA_ERROR();
	// FB 3c
	
	
		updateEtaSO();
		updateBp3();
	}
	CHECK_LAST_CUDA_ERROR();
	updateLAMBDABt1GPU << <_numBlocksM, _blockSize >> > (Bt1._matrixGPU, LAMBDALin._matrixGPU, tradeLin._matrixGPU, _rhog, CoresLinTrans._matrixGPU, _nTrade);
	//Bp3.display();
	CHECK_LAST_CUDA_ERROR();
#ifdef INSTRUMENTATION
	hipDeviceSynchronize();
	t2 = std::chrono::high_resolution_clock::now();
	timePerBlock.increment(0, 5, (float) std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
#endif // INSTRUMENTATION
	
}

void MarEndoConsGPU::updateLocalProbGPU(float epsL, int nIterL)
{
	int numBlocks = _nAgent;
	switch (_blockSize) {
	case 512:
		updateTradePGPUSharedResidual<512> << <numBlocks, _blockSize >> > (Tlocal._matrixGPU, Tlocal_pre._matrixGPU, Tmoy._matrixGPU, P._matrixGPU, MU._matrixGPU, nVoisin._matrixGPU, _at1, _at2, Bt1._matrixGPU, Ct._matrixGPU,
			matLb._matrixGPU, matUb._matrixGPU, Ap1._matrixGPU, Ap3._matrixGPU,  Ap123._matrixGPU, Bp3._matrixGPU, Cp._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, CoresAgentLin._matrixGPU, epsL, nIterL);
		break;
	case 256:
		updateTradePGPUSharedResidual<256> << <numBlocks, _blockSize >> > (Tlocal._matrixGPU, Tlocal_pre._matrixGPU, Tmoy._matrixGPU, P._matrixGPU, MU._matrixGPU, nVoisin._matrixGPU, _at1, _at2, Bt1._matrixGPU, Ct._matrixGPU,
			matLb._matrixGPU, matUb._matrixGPU, Ap1._matrixGPU, Ap3._matrixGPU, Ap123._matrixGPU, Bp3._matrixGPU, Cp._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, CoresAgentLin._matrixGPU, epsL, nIterL);
		break;
	case 128:
		updateTradePGPUSharedResidual<128> << <numBlocks, _blockSize >> > (Tlocal._matrixGPU, Tlocal_pre._matrixGPU, Tmoy._matrixGPU, P._matrixGPU, MU._matrixGPU, nVoisin._matrixGPU, _at1, _at2, Bt1._matrixGPU, Ct._matrixGPU,
			matLb._matrixGPU, matUb._matrixGPU, Ap1._matrixGPU, Ap3._matrixGPU, Ap123._matrixGPU, Bp3._matrixGPU, Cp._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, CoresAgentLin._matrixGPU, epsL, nIterL);
		break;
	case 64:
		updateTradePGPUSharedResidual< 64> << <numBlocks, _blockSize >> > (Tlocal._matrixGPU, Tlocal_pre._matrixGPU, Tmoy._matrixGPU, P._matrixGPU, MU._matrixGPU, nVoisin._matrixGPU, _at1, _at2, Bt1._matrixGPU, Ct._matrixGPU,
			matLb._matrixGPU, matUb._matrixGPU, Ap1._matrixGPU, Ap3._matrixGPU, Ap123._matrixGPU, Bp3._matrixGPU, Cp._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, CoresAgentLin._matrixGPU, epsL, nIterL);
		break;
	case 32:
		updateTradePGPUSharedResidual< 32> << <numBlocks, _blockSize >> > (Tlocal._matrixGPU, Tlocal_pre._matrixGPU, Tmoy._matrixGPU, P._matrixGPU, MU._matrixGPU, nVoisin._matrixGPU, _at1, _at2, Bt1._matrixGPU, Ct._matrixGPU,
			matLb._matrixGPU, matUb._matrixGPU, Ap1._matrixGPU, Ap3._matrixGPU, Ap123._matrixGPU, Bp3._matrixGPU, Cp._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, CoresAgentLin._matrixGPU, epsL, nIterL);
		break;
	case 16:
		updateTradePGPUSharedResidual< 16> << <numBlocks, _blockSize >> > (Tlocal._matrixGPU, Tlocal_pre._matrixGPU, Tmoy._matrixGPU, P._matrixGPU, MU._matrixGPU, nVoisin._matrixGPU, _at1, _at2, Bt1._matrixGPU, Ct._matrixGPU,
			matLb._matrixGPU, matUb._matrixGPU, Ap1._matrixGPU, Ap3._matrixGPU, Ap123._matrixGPU, Bp3._matrixGPU, Cp._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, CoresAgentLin._matrixGPU, epsL, nIterL);
		break;
	case  8:
		updateTradePGPUSharedResidual<  8> << <numBlocks, _blockSize >> > (Tlocal._matrixGPU, Tlocal_pre._matrixGPU, Tmoy._matrixGPU, P._matrixGPU, MU._matrixGPU, nVoisin._matrixGPU, _at1, _at2, Bt1._matrixGPU, Ct._matrixGPU,
			matLb._matrixGPU, matUb._matrixGPU, Ap1._matrixGPU, Ap3._matrixGPU, Ap123._matrixGPU, Bp3._matrixGPU, Cp._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, CoresAgentLin._matrixGPU, epsL, nIterL);
		break;
	case  4:
		updateTradePGPUSharedResidual<  4> << <numBlocks, _blockSize >> > (Tlocal._matrixGPU, Tlocal_pre._matrixGPU, Tmoy._matrixGPU, P._matrixGPU, MU._matrixGPU, nVoisin._matrixGPU, _at1, _at2, Bt1._matrixGPU, Ct._matrixGPU,
			matLb._matrixGPU, matUb._matrixGPU, Ap1._matrixGPU, Ap3._matrixGPU, Ap123._matrixGPU, Bp3._matrixGPU, Cp._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, CoresAgentLin._matrixGPU, epsL, nIterL);
		break;
	case  2:
		updateTradePGPUSharedResidual<  2> << <numBlocks, _blockSize >> > (Tlocal._matrixGPU, Tlocal_pre._matrixGPU, Tmoy._matrixGPU, P._matrixGPU, MU._matrixGPU, nVoisin._matrixGPU, _at1, _at2, Bt1._matrixGPU, Ct._matrixGPU,
			matLb._matrixGPU, matUb._matrixGPU, Ap1._matrixGPU, Ap3._matrixGPU, Ap123._matrixGPU, Bp3._matrixGPU, Cp._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, CoresAgentLin._matrixGPU, epsL, nIterL);
		break;
	case  1:
		updateTradePGPUSharedResidual<  1> << <numBlocks, _blockSize >> > (Tlocal._matrixGPU, Tlocal_pre._matrixGPU, Tmoy._matrixGPU, P._matrixGPU, MU._matrixGPU, nVoisin._matrixGPU, _at1, _at2, Bt1._matrixGPU, Ct._matrixGPU,
			matLb._matrixGPU, matUb._matrixGPU, Ap1._matrixGPU, Ap3._matrixGPU, Ap123._matrixGPU, Bp3._matrixGPU, Cp._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, CoresAgentLin._matrixGPU, epsL, nIterL);
		break;
	}


}


void MarEndoConsGPU::updateEtaSO()
{
	tempN1.subtract(&Pn, &PSO);
	tempN1.multiply(0.5);
	etaSO.add(&tempN1);

	/*for (int n = 0; n < _nAgent; n++) {
		float eta = 0.5 * (Pn.get(n, 0) - PSO.get(n, 0));
		etaSO.set(n, 0, etaSO.get(n, 0) + eta);
	}*/
}



void MarEndoConsGPU::updateBp3()
{
	Bp3.add(&PSO, &Pn);
	Bp3.multiply(0.5);
	Bp3.subtract(&etaSO);
	Bp3.divideT(&nVoisin);
}



float MarEndoConsGPU::updateResBis(int iter)
{
	
	//tradeLin.display();
	float resS = Tlocal.max2(&tradeLin);

	updateDiffGPU << <_numBlocksM, _blockSize >> > (tempNN._matrixGPU, Tlocal._matrixGPU, CoresLinTrans._matrixGPU, _nTrade);
	float resR = tempNN.max2();


	 
	float resXf = PSO.max2(&Pn);
	/*for (int i = 1; i < _nAgentTrue; i++) {
		resXf = MYMAX(abs(PSO.get(i,0) - Pn.get(i,0)), resXf);
		resXf = MYMAX(abs(PSO.get(i + _nAgentTrue, 0) - Pn.get(i + _nAgentTrue, 0)), resXf);
	}*/

	
	
	resF.set(0, iter, resR);
	resF.set(1, iter, resS);
	resF.set(2, iter, resXf);
	return MYMAX(MYMAX(resXf * _ratioEps, resS), resR);
}







void MarEndoConsGPU::display() {

	a.transferCPU();
	b.transferCPU();
	Pn.transferCPU();
	Pmin.transferCPU();
	Pmax.transferCPU();

	if (_iterGlobal == 0) {
		std::cout << "algorithm not launch" << std::endl;
	}
	else if (_iterGlobal < _iterG) {
		std::cout << "method " << _name << " converged in " << _iterGlobal << " iterations." << std::endl;
		std::cout << "Converged in " << (float) timeMarketEndo / CLOCKS_PER_SEC << " seconds" << std::endl;

	}
	else {
		std::cout << "method " << _name << " not converged in " << _iterGlobal << " iterations." << std::endl;
		std::cout << "time taken " << (float) timeMarketEndo / CLOCKS_PER_SEC << " seconds" << std::endl;
	}
	std::cout << "The power error of this state is (constraint) " << resF.get(0, _iterGlobal / _stepG) << " and convergence " << resF.get(1, _iterGlobal / _stepG) << std::endl;
	std::cout << "===============================================================|" << std::endl;
	std::cout << "      System Summary                                           |" << std::endl;
	std::cout << "===============================================================|" << std::endl;
	std::cout << "Agent            " << _nAgentTrue << std::endl;
	


	std::cout << std::endl << std::endl;
	std::cout << std::endl << std::endl;
	std::cout << "========================================================================================================|" << std::endl;
	std::cout << "      Agent Data                                                                                        |" << std::endl;
	std::cout << "========================================================================================================|" << std::endl;
	std::cout << " Agent  |  Cost    |  Cost    |          Power Injection           |           Power Injection          |" << std::endl;
	std::cout << "  #     |   a (pu) |   b (pu) |  P (pu)  | Pmin (pu)  | Pmax (pu)  |  Q (pu)   | Qmin (pu)  | Qmax (pu) |" << std::endl;
	std::cout << "--------|----------|----------|----------|------------|------------|-----------|------------|-----------|" << std::endl;

	for (int n = 0; n < _nAgentTrue; n++) {
		
		std::cout << std::setw(8) << n << "|" << std::setw(9) << a.get(n, 0) << " |" << std::setw(10)
			<< b.get(n, 0) << "|" << std::setw(10) << Pn.get(n, 0) << "|" << std::setw(12)
			<< Pmin.get(n, 0) * nVoisinCPU.get(n, 0) << "|" << std::setw(12) << Pmax.get(n, 0) * nVoisinCPU.get(n, 0)
			<< "|" << std::setw(11) << Pn.get(n + _nAgentTrue, 0) << "|" << std::setw(12) << Pmin.get(n + _nAgentTrue, 0) * nVoisinCPU.get(n + _nAgentTrue, 0)
			<< "|" << std::setw(11) << Pmax.get(n + _nAgentTrue, 0) * nVoisinCPU.get(n + _nAgentTrue, 0) << "|" << std::endl;
	}


	std::cout << "========================================================================================================|" << std::endl;
	std::cout << "                      END PRINT                                                                         |" << std::endl;
	std::cout << "========================================================================================================|" << std::endl;


	a.transferGPU();
	b.transferGPU();
	Pn.transferGPU();
	Pmin.transferGPU();
	Pmax.transferGPU();
}


// updateConsensusGPU << <_numBlocksN, _blockSize >> > (Cost2._matrixGPU, etaSO._matrixGPU, Pn._matrixGPU, Pmarket->_matrixGPU, _rhoSO, _nAgent);
