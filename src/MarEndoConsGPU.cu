#include "hip/hip_runtime.h"
#include "../head/MarEndoConsGPU.cuh"




MarEndoConsGPU::MarEndoConsGPU() : MethodP2PGPU()
{
#if DEBUG_CONSTRUCTOR
	std::cout << " MarEndoConsGPU Constructor" << std::endl;
#endif // DEBUG_CONSTRUCTOR
	_name = NAME;
	timePerBlock = MatrixCPU(1, 9, 0); // Fb0, Fb1 , Fb2, Fb3, Fb5, Fb6 Fb0'
	// si les sous ensemble ne sont pas accessible, tout est dans le premier.
	occurencePerBlock = MatrixCPU(1, 9, 0); //nb de fois utilis� pendant la simu
}


MarEndoConsGPU::MarEndoConsGPU(float rho) : MethodP2PGPU()
{
#if DEBUG_CONSTRUCTOR
	std::cout << "default MarEndoConsGPU Constructor" << std::endl;
#endif // DEBUG_CONSTRUCTOR
	_name = NAME;
	_rho = rho;
	timePerBlock = MatrixCPU(1, 9, 0); // Fb0, Fb1 , Fb2, Fb3, Fb5, Fb6 Fb0'
	// si les sous ensemble ne sont pas accessible, tout est dans le premier.
	occurencePerBlock = MatrixCPU(1, 9, 0); //nb de fois utilis� pendant la simu
}

MarEndoConsGPU::~MarEndoConsGPU()
{
	DELETEB(OPF);
	//DELETEB(OPFCPU);
}


void MarEndoConsGPU::setParam(float rho)
{
	_rho = rho;
}

void MarEndoConsGPU::setTau(float tau)
{
	if (tau < 1) {
		throw std::invalid_argument("tau must be greater than 1");
	}
	_tau = tau;
}



void MarEndoConsGPU::solve(Simparam* result, const Simparam& sim, const StudyCase& cas)
{
#ifdef DEBUG_SOLVE
	cas.display();
	sim.display(1);
#endif // DEBUG_SOLVE
	
	clock_t tall = clock();
#ifdef INSTRUMENTATION
	std::chrono::high_resolution_clock::time_point t1;
	std::chrono::high_resolution_clock::time_point t2;
#endif // INSTRUMENTATION


	// FB 0
	if (_id == 0) {
#ifdef INSTRUMENTATION
		t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION
		init(sim, cas);
#ifdef INSTRUMENTATION
		hipDeviceSynchronize();
		t2 = std::chrono::high_resolution_clock::now();
		timePerBlock.set(0, 0, (float) std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
		occurencePerBlock.set(0, 0, 1);
#endif // INSTRUMENTATION
	}
	
	_rhog = sim.getRho();
	_at1 = _rhog;
	
	
	_resG = 2 * _epsG;
	float epsL2 = _epsL * _epsL;

	_iterGlobal = 0;
	while ((_iterGlobal < _iterG) && (_resG > _epsG)  || (_iterGlobal <=_stepG)) { // || (_iterGlobal <= _stepG)
		//std::cout << "*";
		/*std::cout << "---------------------------------" << std::endl;
		std::cout << " Pn " << std::endl;
		Pn.display(true);
		std::cout << " Pso " << std::endl;
		PSO.display(true);
		std::cout << " Bp3 " << std::endl;
		Bp3.display(true);*/

		/*std::cout << " Tlocal " << std::endl;
		Tlocal.display(true);
		std::cout << " Bt1 " << std::endl;
		Bt1.display(true);
		std::cout << " matlb " << std::endl;
		matLb.display(true);
		std::cout << " matUb " << std::endl;
		matUb.display(true); */
		

#ifdef INSTRUMENTATION
		hipDeviceSynchronize();
		t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION
		//CHECK_LAST_CUDA_ERROR();
		updateLocalProbGPU(epsL2, _iterL);
		//CHECK_LAST_CUDA_ERROR();
#ifdef INSTRUMENTATION
		hipDeviceSynchronize();
		t2 = std::chrono::high_resolution_clock::now();
		timePerBlock.increment(0, 1, (float) std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());

#endif // INSTRUMENTATION
		//std::cout << _iterGlobal << " " << iterLocal << " " << resL << " " << resF.get(0, _iterGlobal / _stepG) << " " << resF.get(1, _iterGlobal / _stepG) << std::endl;

		tradeLin.swap(&Tlocal);
		updateGlobalProb();
		//CHECK_LAST_CUDA_ERROR();
		// FB 4
		if (!(_iterGlobal % _stepG)) {
#ifdef INSTRUMENTATION
			t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION
			_resG = updateResBis( _iterGlobal / _stepG );
			//std::cout << _iterGlobal << " " << iterLocal << " " << resL << " " << resF.get(0, _iterGlobal / _stepG)
				//<< " " << resF.get(1, _iterGlobal / _stepG) << " " << resF.get(2, _iterGlobal / _stepG) << std::endl;
#ifdef INSTRUMENTATION
			hipDeviceSynchronize();
			t2 = std::chrono::high_resolution_clock::now();
			timePerBlock.increment(0, 6, (float) std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
#endif // INSTRUMENTATION		
		}
		//CHECK_LAST_CUDA_ERROR();

		_iterGlobal++;
	}
	
	//std::cout << _iterGlobal << " " << iterLocal  << " " << resF.get(0, (_iterGlobal - 1) / _stepG) << " " << resF.get(1, (_iterGlobal - 1) / _stepG) << " " << resF.get(2, (_iterGlobal - 1) / _stepG) << std::endl;
#ifdef INSTRUMENTATION
	occurencePerBlock.increment(0, 1, _iterGlobal);
	occurencePerBlock.increment(0, 3, _iterGlobal);
	occurencePerBlock.increment(0, 4, _iterGlobal);
	occurencePerBlock.increment(0, 5, _iterGlobal);
	occurencePerBlock.increment(0, 6, _iterGlobal / _stepG);

	t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION
	

	//trade.display();
	/*std::cout << "PSO, Pn" << std::endl;
	PSO.display();
	Pn.display();*/

	// FB 5
	
	MatrixCPU Pb(OPF->getPb());
	MatrixCPU Phi(OPF->getPhi());
	MatrixCPU E(OPF->getE());
	
	result->setE(&E);
	result->setPhi(&Phi);
	result->setPb(&Pb);

	setResult(result, cas.isAC());
	

#ifdef INSTRUMENTATION
	hipDeviceSynchronize();
	t2 = std::chrono::high_resolution_clock::now();
	timePerBlock.increment(0, 7, (float) std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
	occurencePerBlock.increment(0, 7, 1);
	result->setTimeBloc(&timePerBlock, &occurencePerBlock);
#endif // INSTRUMENTATION
}

void MarEndoConsGPU::updateP0(const StudyCase& cas)
{
	_id = _id + 1;
#ifdef INSTRUMENTATION
	std::chrono::high_resolution_clock::time_point t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION
	
	matLb.transferCPU();
	matUb.transferCPU();

	Pmin = cas.getPmin();
	Pmax = cas.getPmax();


	MatrixCPU Lb(cas.getLb());
	MatrixCPU Ub(cas.getUb());

	b = cas.getb();
	Cp = b;
	int indice = 0;

	for (int idAgent = 0; idAgent < _nAgent; idAgent++) {
		int Nvoisinmax = nVoisinCPU.get(idAgent, 0);
		for (int voisin = 0; voisin < Nvoisinmax; voisin++) {
			matLb.set(indice, 0, Lb.get(idAgent, 0));
			matUb.set(indice, 0, Ub.get(idAgent, 0));
			indice = indice + 1;
		}
	}
	matLb.transferGPU();
	matUb.transferGPU();

	float Ploss = Pn.get(0, 0, false);
	float Qloss = Pn.get(_nAgentTrue, 0, false);
	// pour essayer que cela marche
	Pn.add(&Pmin, &Pmax);
	Pn.divide(2);

	// unleash powe
	Pmin.set(0, 0, -POWERLIMIT, true);
	Pmax.set(_nAgentTrue, 0, POWERLIMIT, true);
	Pmin.set(_nAgentTrue, 0, -POWERLIMIT, true);

	Pn.set(0, 0, Ploss, true);
	Pn.set(_nAgentTrue, 0, Qloss, true);

	Pmin.divideT(&nVoisin);
	Pmax.divideT(&nVoisin);
	Cp.multiplyT(&nVoisin);
#ifdef INSTRUMENTATION
	hipDeviceSynchronize();
	std::chrono::high_resolution_clock::time_point t2 = std::chrono::high_resolution_clock::now();
	timePerBlock.increment(0, 8, (float) std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
	occurencePerBlock.increment(0, 8, 1);
#endif // INSTRUMENTATION


}

void MarEndoConsGPU::init(const Simparam& sim, const StudyCase& cas)
{
	DELETEB(OPF);
	DELETEB(OPFCPU);
	//std::cout << "init " << std::endl;
	
	if (!cas.isAC()) {
		throw std::invalid_argument("Wrong studyCase must be AC");
	}
	if (!cas.isRadial()) {
		throw std::invalid_argument("Wrong studyCase must be radial, dont have OPF on non-radial cases");
	}
	////CHECK_LAST_CUDA_ERROR();
	isAC = true;
	initSize(cas);
	if (_nAgentTrue != sim.getNAgent()) {
		throw std::invalid_argument("nAgent different on Simparam and study case");
	}
	initSimParam(sim);

	_rhoSO = _rhog; // _rhoSO = _rho1;
	paramOPF = sim;
	paramOPF.setItG(_iterIntern);
	paramOPF.setEpsG(_epsIntern);
	
	
	int nVoisinMax = nVoisin.max2();
	if (_blockSize * NMAXPEERPERTRHREAD < nVoisinMax) {
		std::cout << _blockSize << " " << NMAXPEERPERTRHREAD << " " << nVoisinMax << std::endl;
		throw std::invalid_argument("For this Method, an agent must not have more than 5120 peers");
	}
	//std::cout << "Market" << std::endl;
		
	initCaseParam(sim, cas);
	Pmin.set(0, 0, -100000, true); // unleash power !!!
	Pmin.set(_nAgentTrue, 0, -100000, true); // unleash power !!!	
	Pmax.set(_nAgentTrue, 0, 100000, true); // unleash power !!!

	if (initWithMarketClear) {
		std::chrono::high_resolution_clock::time_point t1 = std::chrono::high_resolution_clock::now();

		ADMMMarketGPU market;
		Simparam res(sim);
		market.solve(&res, sim, cas);
		//res.display();
		LAMBDA = res.getLambda();
		trade = res.getTrade();
		Pn = res.getPn();
		Tmoy = Pn;
		std::chrono::high_resolution_clock::time_point t2 = std::chrono::high_resolution_clock::now();
		//std::cout << "time : " <<  (float)std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count() / 1000000000 << std::endl;
	}
	MatrixCPU PnCPU;
	Pn.toMatCPU(PnCPU);
	//Pn.display(true);
	paramOPF.setPn(&PnCPU);
	
	//std::cout << "mise sous forme lineaire" << std::endl;
	initLinForm(cas);
	
	//CHECK_LAST_CUDA_ERROR();
	//std::cout << "donnees sur CPU pour le grid" << std::endl;
	

	PSO = MatrixGPU(_nAgent, 1, 0, 1);
	PSO.preallocateReduction();
	etaSO = MatrixGPU(_nAgent, 1, 0, 1);
	Bp3 = MatrixGPU(_nAgent, 1, 0, 1);
	
	if (OPFonCPU) {
		OPFCPU = new OPFADMMCons;
		OPFCPU->initConsensus(paramOPF, cas, _rhoSO);
	}
	else {
		OPF = new OPFADMMConsGPU;
		OPF->initConsensus(paramOPF, cas, _rhoSO);
	}
	CHECK_LAST_CUDA_ERROR();
	//std::cout << "autres donnee sur CPU" << std::endl;
	
	initP2PMarket();
	
	// on en veut pas que l'agent des pertes consomme plus que n�cessaire !!!
	//a.set(0, 0, 1);
	//a.set(_nAgentTrue, 0, 1);

	//CHECK_LAST_CUDA_ERROR();
	Ap3 = nVoisin;
	Ap3.multiplyT(&nVoisin);
	Ap3.multiply(_rhoSO);
	Ap123.add(&Ap12, &Ap3);	
	P = Tmoy;
	
	updateGlobalProb();
	
	//std::cout << "fin init temps : " << (float)(clock() - t) / CLOCKS_PER_SEC << std::endl;
	//std::cout << "fin init " << std::endl;
}

void MarEndoConsGPU::updateGlobalProb() {

	// FB 3a
#ifdef INSTRUMENTATION
	std::chrono::high_resolution_clock::time_point t1 = std::chrono::high_resolution_clock::now();
	std::chrono::high_resolution_clock::time_point t2;
#endif // INSTRUMENTATION

	
	float eps = MYMIN(_resG * _delta, _epsIntern);
	
	
	//std::cout << "SolveOPF" << std::endl;
	if (_iterGlobal % _stepIntern == 0) {
		if (OPFonCPU) {
			PSO.toMatCPU(PSOCPU);
			OPFCPU->solveConsensus(eps, &PSOCPU);
			PSO = PSOCPU;
		}
		else {
			OPF->solveConsensus(eps, &PSO);
		}
		CHECK_LAST_CUDA_ERROR();
	
#ifdef INSTRUMENTATION
	hipDeviceSynchronize();
	t2 = std::chrono::high_resolution_clock::now();
	timePerBlock.increment(0, 4, (float) std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
	// FB 3b
	t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION
	//std::cout << "update OPF" << std::endl;

	updatePn();
	CHECK_LAST_CUDA_ERROR();
	if (OPFonCPU) {
		Pn.toMatCPU(PnCPU);
		OPFCPU->updateConsensus(&PnCPU);
	}
	else {
		OPF->updateConsensus(&Pn);
	}
	CHECK_LAST_CUDA_ERROR();
	

#ifdef INSTRUMENTATION
	hipDeviceSynchronize();
	t2 = std::chrono::high_resolution_clock::now();
	timePerBlock.increment(0, 3, (float) std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
	
	t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION
	//Agent des pertes
	//std::cout << "update Market" << std::endl;
	/*Pmin.set(0, 0, Ploss / nVoisin.get(0,0));
	Pmax.set(0, 0, Ploss / nVoisin.get(0, 0));
	
	Pmin.set(_nAgentTrue, 0, Qloss / nVoisin.get(_nAgentTrue, 0));
	Pmax.set(_nAgentTrue, 0, Qloss / nVoisin.get(_nAgentTrue, 0));*/
	

	float Qloss = PSO.get(_nAgentTrue, 0, false);
	CHECK_LAST_CUDA_ERROR();
	if (Qloss > 0) {
		matUb.setBloc(_nTradeP, _nTradeP + _nAgentTrue - 1, 0, 1, Qloss);
		matLb.setBloc(_nTradeP, _nTradeP + _nAgentTrue - 1, 0, 1, 0.0);
	}
	else {
		matLb.setBloc(_nTradeP, _nTradeP + _nAgentTrue - 1, 0, 1, Qloss);
		matUb.setBloc(_nTradeP, _nTradeP + _nAgentTrue - 1, 0, 1, 0.0);
	}
	/**/
	CHECK_LAST_CUDA_ERROR();
	// FB 3c
	
	
		updateEtaSO();
		updateBp3();
	}
	CHECK_LAST_CUDA_ERROR();
	updateLAMBDABt1GPU << <_numBlocksM, _blockSize >> > (Bt1._matrixGPU, LAMBDALin._matrixGPU, tradeLin._matrixGPU, _rhog, CoresLinTrans._matrixGPU, _nTrade);
	//Bp3.display();
	CHECK_LAST_CUDA_ERROR();
#ifdef INSTRUMENTATION
	hipDeviceSynchronize();
	t2 = std::chrono::high_resolution_clock::now();
	timePerBlock.increment(0, 5, (float) std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
#endif // INSTRUMENTATION
	
}

void MarEndoConsGPU::updateLocalProbGPU(float epsL, int nIterL)
{
	int numBlocks = _nAgent;
	switch (_blockSize) {
	case 512:
		updateTradePGPUSharedResidual<512> << <numBlocks, _blockSize >> > (Tlocal._matrixGPU, Tlocal_pre._matrixGPU, Tmoy._matrixGPU, P._matrixGPU, MU._matrixGPU, nVoisin._matrixGPU, _at1, _at2, Bt1._matrixGPU, Ct._matrixGPU,
			matLb._matrixGPU, matUb._matrixGPU, Ap1._matrixGPU, Ap3._matrixGPU,  Ap123._matrixGPU, Bp3._matrixGPU, Cp._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, CoresAgentLin._matrixGPU, epsL, nIterL);
		break;
	case 256:
		updateTradePGPUSharedResidual<256> << <numBlocks, _blockSize >> > (Tlocal._matrixGPU, Tlocal_pre._matrixGPU, Tmoy._matrixGPU, P._matrixGPU, MU._matrixGPU, nVoisin._matrixGPU, _at1, _at2, Bt1._matrixGPU, Ct._matrixGPU,
			matLb._matrixGPU, matUb._matrixGPU, Ap1._matrixGPU, Ap3._matrixGPU, Ap123._matrixGPU, Bp3._matrixGPU, Cp._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, CoresAgentLin._matrixGPU, epsL, nIterL);
		break;
	case 128:
		updateTradePGPUSharedResidual<128> << <numBlocks, _blockSize >> > (Tlocal._matrixGPU, Tlocal_pre._matrixGPU, Tmoy._matrixGPU, P._matrixGPU, MU._matrixGPU, nVoisin._matrixGPU, _at1, _at2, Bt1._matrixGPU, Ct._matrixGPU,
			matLb._matrixGPU, matUb._matrixGPU, Ap1._matrixGPU, Ap3._matrixGPU, Ap123._matrixGPU, Bp3._matrixGPU, Cp._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, CoresAgentLin._matrixGPU, epsL, nIterL);
		break;
	case 64:
		updateTradePGPUSharedResidual< 64> << <numBlocks, _blockSize >> > (Tlocal._matrixGPU, Tlocal_pre._matrixGPU, Tmoy._matrixGPU, P._matrixGPU, MU._matrixGPU, nVoisin._matrixGPU, _at1, _at2, Bt1._matrixGPU, Ct._matrixGPU,
			matLb._matrixGPU, matUb._matrixGPU, Ap1._matrixGPU, Ap3._matrixGPU, Ap123._matrixGPU, Bp3._matrixGPU, Cp._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, CoresAgentLin._matrixGPU, epsL, nIterL);
		break;
	case 32:
		updateTradePGPUSharedResidual< 32> << <numBlocks, _blockSize >> > (Tlocal._matrixGPU, Tlocal_pre._matrixGPU, Tmoy._matrixGPU, P._matrixGPU, MU._matrixGPU, nVoisin._matrixGPU, _at1, _at2, Bt1._matrixGPU, Ct._matrixGPU,
			matLb._matrixGPU, matUb._matrixGPU, Ap1._matrixGPU, Ap3._matrixGPU, Ap123._matrixGPU, Bp3._matrixGPU, Cp._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, CoresAgentLin._matrixGPU, epsL, nIterL);
		break;
	case 16:
		updateTradePGPUSharedResidual< 16> << <numBlocks, _blockSize >> > (Tlocal._matrixGPU, Tlocal_pre._matrixGPU, Tmoy._matrixGPU, P._matrixGPU, MU._matrixGPU, nVoisin._matrixGPU, _at1, _at2, Bt1._matrixGPU, Ct._matrixGPU,
			matLb._matrixGPU, matUb._matrixGPU, Ap1._matrixGPU, Ap3._matrixGPU, Ap123._matrixGPU, Bp3._matrixGPU, Cp._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, CoresAgentLin._matrixGPU, epsL, nIterL);
		break;
	case  8:
		updateTradePGPUSharedResidual<  8> << <numBlocks, _blockSize >> > (Tlocal._matrixGPU, Tlocal_pre._matrixGPU, Tmoy._matrixGPU, P._matrixGPU, MU._matrixGPU, nVoisin._matrixGPU, _at1, _at2, Bt1._matrixGPU, Ct._matrixGPU,
			matLb._matrixGPU, matUb._matrixGPU, Ap1._matrixGPU, Ap3._matrixGPU, Ap123._matrixGPU, Bp3._matrixGPU, Cp._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, CoresAgentLin._matrixGPU, epsL, nIterL);
		break;
	case  4:
		updateTradePGPUSharedResidual<  4> << <numBlocks, _blockSize >> > (Tlocal._matrixGPU, Tlocal_pre._matrixGPU, Tmoy._matrixGPU, P._matrixGPU, MU._matrixGPU, nVoisin._matrixGPU, _at1, _at2, Bt1._matrixGPU, Ct._matrixGPU,
			matLb._matrixGPU, matUb._matrixGPU, Ap1._matrixGPU, Ap3._matrixGPU, Ap123._matrixGPU, Bp3._matrixGPU, Cp._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, CoresAgentLin._matrixGPU, epsL, nIterL);
		break;
	case  2:
		updateTradePGPUSharedResidual<  2> << <numBlocks, _blockSize >> > (Tlocal._matrixGPU, Tlocal_pre._matrixGPU, Tmoy._matrixGPU, P._matrixGPU, MU._matrixGPU, nVoisin._matrixGPU, _at1, _at2, Bt1._matrixGPU, Ct._matrixGPU,
			matLb._matrixGPU, matUb._matrixGPU, Ap1._matrixGPU, Ap3._matrixGPU, Ap123._matrixGPU, Bp3._matrixGPU, Cp._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, CoresAgentLin._matrixGPU, epsL, nIterL);
		break;
	case  1:
		updateTradePGPUSharedResidual<  1> << <numBlocks, _blockSize >> > (Tlocal._matrixGPU, Tlocal_pre._matrixGPU, Tmoy._matrixGPU, P._matrixGPU, MU._matrixGPU, nVoisin._matrixGPU, _at1, _at2, Bt1._matrixGPU, Ct._matrixGPU,
			matLb._matrixGPU, matUb._matrixGPU, Ap1._matrixGPU, Ap3._matrixGPU, Ap123._matrixGPU, Bp3._matrixGPU, Cp._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, CoresAgentLin._matrixGPU, epsL, nIterL);
		break;
	}
}


void MarEndoConsGPU::updateEtaSO()
{
	tempN1.subtract(&Pn, &PSO);
	tempN1.multiply(0.5);
	etaSO.add(&tempN1);

	/*for (int n = 0; n < _nAgent; n++) {
		float eta = 0.5 * (Pn.get(n, 0) - PSO.get(n, 0));
		etaSO.set(n, 0, etaSO.get(n, 0) + eta);
	}*/
}



void MarEndoConsGPU::updateBp3()
{
	Bp3.add(&PSO, &Pn);
	Bp3.multiply(0.5);
	Bp3.subtract(&etaSO);
	Bp3.divideT(&nVoisin);
}



float MarEndoConsGPU::updateResBis(int iter)
{
	
	//tradeLin.display();
	float resS = Tlocal.max2(&tradeLin);

	updateDiffGPU << <_numBlocksM, _blockSize >> > (tempNN._matrixGPU, Tlocal._matrixGPU, CoresLinTrans._matrixGPU, _nTrade);
	float resR = tempNN.max2();


	 
	float resXf = PSO.max2(&Pn);
	/*for (int i = 1; i < _nAgentTrue; i++) {
		resXf = MYMAX(abs(PSO.get(i,0) - Pn.get(i,0)), resXf);
		resXf = MYMAX(abs(PSO.get(i + _nAgentTrue, 0) - Pn.get(i + _nAgentTrue, 0)), resXf);
	}*/

	
	
	resF.set(0, iter, resR);
	resF.set(1, iter, resS);
	resF.set(2, iter, resXf);
	return MYMAX(MYMAX(resXf * _ratioEps, resS), resR);
}







void MarEndoConsGPU::display() {

	a.transferCPU();
	b.transferCPU();
	Pn.transferCPU();
	Pmin.transferCPU();
	Pmax.transferCPU();

	if (_iterGlobal == 0) {
		std::cout << "algorithm not launch" << std::endl;
	}
	else if (_iterGlobal < _iterG) {
		std::cout << "method " << _name << " converged in " << _iterGlobal << " iterations." << std::endl;
		std::cout << "Converged in " << (float) tMarket / CLOCKS_PER_SEC << " seconds" << std::endl;

	}
	else {
		std::cout << "method " << _name << " not converged in " << _iterGlobal << " iterations." << std::endl;
		std::cout << "time taken " << (float) tMarket / CLOCKS_PER_SEC << " seconds" << std::endl;
	}
	std::cout << "The power error of this state is (constraint) " << resF.get(0, _iterGlobal / _stepG) << " and convergence " << resF.get(1, _iterGlobal / _stepG) << std::endl;
	std::cout << "===============================================================|" << std::endl;
	std::cout << "      System Summary                                           |" << std::endl;
	std::cout << "===============================================================|" << std::endl;
	std::cout << "Agent            " << _nAgentTrue << std::endl;
	


	std::cout << std::endl << std::endl;
	std::cout << std::endl << std::endl;
	std::cout << "========================================================================================================|" << std::endl;
	std::cout << "      Agent Data                                                                                        |" << std::endl;
	std::cout << "========================================================================================================|" << std::endl;
	std::cout << " Agent  |  Cost    |  Cost    |          Power Injection           |           Power Injection          |" << std::endl;
	std::cout << "  #     |   a (pu) |   b (pu) |  P (pu)  | Pmin (pu)  | Pmax (pu)  |  Q (pu)   | Qmin (pu)  | Qmax (pu) |" << std::endl;
	std::cout << "--------|----------|----------|----------|------------|------------|-----------|------------|-----------|" << std::endl;

	for (int n = 0; n < _nAgentTrue; n++) {
		
		std::cout << std::setw(8) << n << "|" << std::setw(9) << a.get(n, 0) << " |" << std::setw(10)
			<< b.get(n, 0) << "|" << std::setw(10) << Pn.get(n, 0) << "|" << std::setw(12)
			<< Pmin.get(n, 0) * nVoisinCPU.get(n, 0) << "|" << std::setw(12) << Pmax.get(n, 0) * nVoisinCPU.get(n, 0)
			<< "|" << std::setw(11) << Pn.get(n + _nAgentTrue, 0) << "|" << std::setw(12) << Pmin.get(n + _nAgentTrue, 0) * nVoisinCPU.get(n + _nAgentTrue, 0)
			<< "|" << std::setw(11) << Pmax.get(n + _nAgentTrue, 0) * nVoisinCPU.get(n + _nAgentTrue, 0) << "|" << std::endl;
	}


	std::cout << "========================================================================================================|" << std::endl;
	std::cout << "                      END PRINT                                                                         |" << std::endl;
	std::cout << "========================================================================================================|" << std::endl;


	a.transferGPU();
	b.transferGPU();
	Pn.transferGPU();
	Pmin.transferGPU();
	Pmax.transferGPU();
}


// updateConsensusGPU << <_numBlocksN, _blockSize >> > (Cost2._matrixGPU, etaSO._matrixGPU, Pn._matrixGPU, Pmarket->_matrixGPU, _rhoSO, _nAgent);
