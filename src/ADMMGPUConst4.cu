#include "hip/hip_runtime.h"
#include "../head/ADMMGPUConst4.cuh"

ADMMGPUConst4::ADMMGPUConst4() : MethodP2PGPU()
{
#if DEBUG_CONSTRUCTOR
	std::cout << "Constructeur ADMMGPUConst4" << std::endl;
#endif // DEBUG_CONSTRUCTOR
	_name = NAME;
}


ADMMGPUConst4::ADMMGPUConst4(float rho) : MethodP2PGPU()
{
#if DEBUG_CONSTRUCTOR
	std::cout << "Constructeur ADMMGPUConst4 defaut" << std::endl;
#endif // DEBUG_CONSTRUCTOR
	_name = NAME;
	_rho = rho;
}

ADMMGPUConst4::~ADMMGPUConst4()
{
}

void ADMMGPUConst4::setParam(float rho)
{
	_rho = rho;
}

void ADMMGPUConst4::setTau(float tau)
{
	if (tau < 1) {
		throw std::invalid_argument("tau must be greater than 1");
	}
	_tau = tau;
}

void ADMMGPUConst4::init(const Simparam& sim, const StudyCase& cas)
{
	// intitilisation des matrixs et variables 
	clock_t t = clock();
	_rhog = sim.getRho();
	_rho1 = sim.getRho1();
	//std::cout << "rho initial " << _rhog << std::endl;
	_nAgent = sim.getNAgent();
	
	_rhol = _rho;
	if (_rho == 0) {
		_rhol = _rhog;
	}
	const int iterG = sim.getIterG();
	const int stepG = sim.getStepG();
	float epsG = sim.getEpsG();
	float epsGC = sim.getEpsGC();
	_ratioEps = epsG / epsGC;
	
	if (cas.isAC()) {
		MatrixCPU nVoisinT = cas.getNvoi();
		nVoisinCPU = MatrixCPU(_nAgent, 1);
		for (int n = 0; n < _nAgent; n++) {
			nVoisinCPU.set(n, 0, nVoisinT.get(n, 0));
		}
	}
	else {
		nVoisinCPU = cas.getNvoi();
	}
	
	nVoisin = MatrixGPU(nVoisinCPU, 1);
	nVoisin.preallocateReduction();
	
	int nVoisinMax = nVoisin.max2();
	if (_blockSize * NMAXPEERPERTRHREAD < nVoisinMax) {
		std::cout << _blockSize << " " << NMAXPEERPERTRHREAD << " " << nVoisinMax << std::endl;
		throw std::invalid_argument("For this Method, an agent must not have more than 5120 peers");
	}

	_nLine = cas.getNLine();
	
	_nBus = cas.getNBus();

	_nTrade = nVoisin.sum();
	//std::cout << "nVoisinma " << nVoisinMax << "trade " << _nTrade << std::endl;
	_numBlocksN = ceil((_nAgent + _blockSize - 1) / _blockSize);
	_numBlocksM = ceil((_nTrade + _blockSize - 1) / _blockSize);
	_numBlocksL = ceil((_nLine + _blockSize - 1) / _blockSize);
	_numBlocksNL = ceil((_nAgent * _nLine + _blockSize - 1) / _blockSize);
	_at1 = _rhog; // represente en fait 2*a
	_at2 = _rhol;

	resF = MatrixCPU(3, (iterG / stepG) + 1);
	resX = MatrixCPU(4, (iterG / stepG) + 1);

	MatrixCPU BETA(cas.getBeta());
	MatrixGPU Ub(cas.getUb());
	MatrixGPU Lb(cas.getLb());
	LAMBDA = sim.getLambda();
	trade = sim.getTrade();
	
	//std::cout << "mise sous forme lin�aire" << std::endl;
	// Rem : si matrice d�j� existante, elles sont d�j� sur GPU donc bug pour les get
	if (Ct.getPos()) { // une copie en trop mais pour l'instant c'est ok...
		CoresMatLin.transferCPU();

		CoresLinAgent.transferCPU();
		CoresAgentLin.transferCPU();
		CoresLinVoisin.transferCPU();
		CoresLinTrans.transferCPU();

		Tlocal_pre.transferCPU();
		tradeLin.transferCPU();
		LAMBDALin.transferCPU();

		matLb.transferCPU();
		matUb.transferCPU();
		Ct.transferCPU();

		if (cas.isAC()) {
			a.transferCPU();
			b.transferCPU();
			Pmin.transferCPU();
			Pmax.transferCPU();
			MU.transferCPU();
			Tmoy.transferCPU();
		}
	}


	CoresMatLin = MatrixGPU(_nAgent, _nAgent, -1);
	CoresLinAgent = MatrixGPU(_nTrade, 1);
	CoresAgentLin = MatrixGPU(_nAgent + 1, 1);
	CoresLinVoisin = MatrixGPU(_nTrade, 1);
	CoresLinTrans = MatrixGPU(_nTrade, 1);

	Tlocal_pre = MatrixGPU(_nTrade, 1);
	tradeLin = MatrixGPU(_nTrade, 1);
	LAMBDALin = MatrixGPU(_nTrade, 1);

	matLb = MatrixGPU(_nTrade, 1);
	matUb = MatrixGPU(_nTrade, 1);
	Ct = MatrixGPU(_nTrade, 1);

	int indice = 0;

	for (int idAgent = 0; idAgent < _nAgent; idAgent++) {
		MatrixCPU omega(cas.getVoisin(idAgent));
		int Nvoisinmax = nVoisinCPU.get(idAgent, 0);
		for (int voisin = 0; voisin < Nvoisinmax; voisin++) {
			int idVoisin = omega.get(voisin, 0);
			if(Lb.getNCol()==1){
				matLb.set(indice, 0, Lb.get(idAgent, 0));
				matUb.set(indice, 0, Ub.get(idAgent, 0));
			} else {
				matLb.set(indice, 0, Lb.get(idAgent, idVoisin));
				matUb.set(indice, 0, Ub.get(idAgent, idVoisin));
			}
			Ct.set(indice, 0, BETA.get(idAgent, idVoisin));
			tradeLin.set(indice, 0, trade.get(idAgent, idVoisin));
			Tlocal_pre.set(indice, 0, trade.get(idAgent, idVoisin));
			LAMBDALin.set(indice, 0, LAMBDA.get(idAgent, idVoisin));
			CoresLinAgent.set(indice, 0, idAgent);
			CoresLinVoisin.set(indice, 0, idVoisin);
			CoresMatLin.set(idAgent, idVoisin, indice);
			indice = indice + 1;
		}
		CoresAgentLin.set(idAgent + 1, 0, indice);
	}
	for (int lin = 0; lin < _nTrade; lin++) {
		int i = CoresLinAgent.get(lin, 0);
		int j = CoresLinVoisin.get(lin, 0);
		int k = CoresMatLin.get(j, i);
		CoresLinTrans.set(lin, 0, k);
	}


	// transfert des mises lineaire
	matUb.transferGPU();
	matLb.transferGPU();
	Ct.transferGPU();

	Tlocal_pre.transferGPU();
	tradeLin.transferGPU();
	LAMBDALin.transferGPU();

	CoresAgentLin.transferGPU();
	CoresLinAgent.transferGPU();
	CoresLinVoisin.transferGPU();
	CoresMatLin.transferGPU();
	CoresLinTrans.transferGPU();

	//std::cout << "donnees sur GPU pour le grid" << std::endl;
	Kappa1 = MatrixGPU(_nLine, 1, 0, 1);
	Kappa2 = MatrixGPU(_nLine, 1, 0, 1);
	Kappa1_pre = MatrixGPU(_nLine, 1, 0, 1);
	Kappa2_pre = MatrixGPU(_nLine, 1, 0, 1);
	Qpart = MatrixGPU(_nAgent, _nLine, 0, 1);
	Qtot = MatrixGPU(_nLine, 1, 0, 1);
	alpha = MatrixGPU(_nAgent, _nLine, 0, 1);
	G = MatrixGPU(cas.getPowerSensi());

	lLimit = MatrixGPU(cas.getLineLimit(), 1);

	GTrans = MatrixGPU(_nAgent, _nLine);

	if (GTrans.getPos()) {
		GTrans.transferCPU();
	}

	GTrans.setTrans(&G);
	//G.transferGPU();

	GTrans.transferGPU();

	G2 = GTrans;

	G2.multiplyT(&GTrans);

	//std::cout << "autres donn�e sur GPU" << std::endl;
	tempNN = MatrixGPU(_nTrade, 1, 0, 1);
	tempN1 = MatrixGPU(_nAgent, 1, 0, 1); // plut�t que de re-allouer de la m�moire � chaque utilisation
	tempL1 = MatrixGPU(_nLine, 1, 0, 1);
	tempL2 = MatrixGPU(_nLine, 1, 0, 1);
	//MatrixGPU temp1N(1, _nAgent, 0, 1);

	Tlocal = MatrixGPU(_nTrade, 1, 0, 1);
	P = MatrixGPU(_nAgent, 1, 0, 1); // moyenne des trades
	Pn = MatrixGPU(_nAgent, 1, 0, 1); // somme des trades


	if (cas.isAC()) {
		MatrixCPU aT = cas.geta();
		MatrixCPU bT = cas.getb();
		MatrixCPU PminT = cas.getPmin();
		MatrixCPU PmaxT = cas.getPmax();
		MatrixCPU MUT = sim.getMU(); // facteur reduit i.e lambda_l/_rho
		MatrixCPU TmoyT = sim.getPn();
		a = MatrixGPU(_nAgent, 1);
		b = MatrixGPU(_nAgent, 1);
		Pmin = MatrixGPU(_nAgent, 1);
		Pmax = MatrixGPU(_nAgent, 1);
		MU = MatrixGPU(_nAgent, 1);
		Tmoy = MatrixGPU(_nAgent, 1);

		for (int n = 0; n < _nAgent; n++) {
			a.set(n, 0, aT.get(n, 0));
			b.set(n, 0, bT.get(n, 0));
			Pmin.set(n, 0, PminT.get(n, 0));
			Pmax.set(n, 0, PmaxT.get(n, 0));
			MU.set(n, 0, MUT.get(n, 0));
			Tmoy.set(n, 0, TmoyT.get(n, 0));
		}
		a.transferGPU();
		b.transferGPU();
		Pmin.transferGPU();
		Pmax.transferGPU();
		MU.transferGPU();
		Tmoy.transferGPU();
	}
	else {
		a = MatrixGPU(cas.geta(), 1);
		b = MatrixGPU(cas.getb(), 1);

		Pmin = MatrixGPU(cas.getPmin(), 1);
		Pmax = MatrixGPU(cas.getPmax(), 1);
		MU = MatrixGPU(sim.getMU(), 1); // facteur reduit i.e lambda_l/_rho
		Tmoy = MatrixGPU(sim.getPn(), 1);
	}
	Ap1 = nVoisin;
	Ap2 = a;
	Cp1 = b;

	Ap12 = MatrixGPU(_nAgent, 1, 0, 1);
	Ap2a = a;
	Ap2b = MatrixGPU(_nAgent, 1, 0, 1);

	Bt1 = MatrixGPU(_nTrade, 1, 0, 1);
	Cp = MatrixGPU(_nAgent, 1, 0, 1);
	Cp2 = MatrixGPU(_nAgent, 1, 0, 1);
	Cp1 = b;

	

	CHECK_LAST_CUDA_ERROR();
	tempNN.preallocateReduction();
	Tlocal.preallocateReduction();
	tempL1.preallocateReduction();

	P.preallocateReduction();

	Pmin.divideT(&nVoisin);
	Pmax.divideT(&nVoisin);
	Ap1.multiply(_rhol);
	Cp1.multiplyT(&nVoisin);
	Tmoy.divideT(&nVoisin);


	tempN1.sum(&G2);
	tempN1.multiply(2 * _rho1);
	Ap2.add(&tempN1);
	Ap2.multiplyT(&nVoisin);
	Ap2.multiplyT(&nVoisin);
	Ap12.add(&Ap1, &Ap2);
	
	/*Ap2b.sum(&G2);
	Ap2b.multiply(2 * _rho1);
	Ap2a.multiplyT(&nVoisin);
	Ap2a.multiplyT(&nVoisin);
	Ap2b.multiplyT(&nVoisin);
	Ap2b.multiplyT(&nVoisin);

	Ap2.add(&Ap2a, &Ap2b);*/

	updateGlobalProbGPU();
	//std::cout << "rho " << _rhog << " rhoL " << _rhol << " _rho1 " << _rho1 << std::endl;
	//std::cout << "fin init temps : " << (float)(clock() - t) / CLOCKS_PER_SEC << std::endl;
	
}



void ADMMGPUConst4::updateP0(const StudyCase& cas)
{
	_id = _id + 1;
#ifdef INSTRUMENTATION
	hipDeviceSynchronize();
	std::chrono::high_resolution_clock::time_point t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION
	

	matLb.transferCPU();
	
	b.transferCPU();
	Pmin.transferCPU();
	Pmax.transferCPU();
	if (cas.isAC()) {
		
		MatrixCPU bT = cas.getb();
		MatrixCPU PminT = cas.getPmin();
		MatrixCPU PmaxT = cas.getPmax();


		for (int n = 0; n < _nAgent; n++) {
			
			b.set(n, 0, bT.get(n, 0));
			Pmin.set(n, 0, PminT.get(n, 0));
			Pmax.set(n, 0, PmaxT.get(n, 0));
		}
		
		b.transferGPU();
		Pmin.transferGPU();
		Pmax.transferGPU();
	}
	else {
	
		b = cas.getb();
		Pmin = cas.getPmin();
		Pmax = cas.getPmax();
		
	}


	


	MatrixGPU Lb(cas.getLb());

	Cp1 = b;
	int indice = 0;

	for (int idAgent = 0; idAgent < _nAgent; idAgent++) {
		int Nvoisinmax = nVoisinCPU.get(idAgent, 0);
		for (int voisin = 0; voisin < Nvoisinmax; voisin++) {
			matLb.set(indice, 0, Lb.get(idAgent, 0));
			indice = indice + 1;
		}
	}

	
	matLb.transferGPU();

	Pmin.divideT(&nVoisin);
	Pmax.divideT(&nVoisin);
	Cp1.multiplyT(&nVoisin);
#ifdef INSTRUMENTATION
	hipDeviceSynchronize();
	std::chrono::high_resolution_clock::time_point t2 = std::chrono::high_resolution_clock::now();
	timePerBlock.increment(0, 8, (float) std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
	occurencePerBlock.increment(0, 8, 1);
#endif // INSTRUMENTATION

}


void ADMMGPUConst4::solve(Simparam* result, const Simparam& sim, const StudyCase& cas)
{
#ifdef DEBUG_SOLVE
	cas.display();
	sim.display(1);
#endif // DEBUG_SOLVE
	
	
	clock_t tall = clock();

#ifdef INSTRUMENTATION
	std::chrono::high_resolution_clock::time_point t1;
	std::chrono::high_resolution_clock::time_point t2;	
#endif // INSTRUMENTATION




	if (_id == 0) {
#ifdef INSTRUMENTATION
		hipDeviceSynchronize();
		t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION
		init(sim, cas);
#ifdef INSTRUMENTATION
		hipDeviceSynchronize();
		t2 = std::chrono::high_resolution_clock::now();
		timePerBlock.increment(0, 0, (float) std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
		occurencePerBlock.increment(0, 0, 1);
#endif // INSTRUMENTATION
	}
	_rhog = sim.getRho();
	//std::cout << _numBlocks2 << " " <<  _blockSize << std::endl;
	/*_mu = _muInit;
	_mu1 = _muInit;
	_rhog = sim.getRho();
	float oldRho1 = _rho1;
	_rho1 = sim.getRho1();
	float tau = _rho1 / oldRho1;
	Ap2b.multiply(_tau);
	Ap2.add(&Ap2a, &Ap2b);
	Ap12.add(&Ap1, &Ap2);*/

	_at1 = _rhog; // represente en fait 2*a
	
	float epsG = sim.getEpsG();
	float epsL = sim.getEpsL();
	const int stepL = sim.getStepL();
	const int stepG = sim.getStepG();
	const int iterG = sim.getIterG();
	const int iterL = sim.getIterL();
	

	float resG = 2 * epsG;
	float epsL2 = epsL * epsL;
	int iterGlobal = 0;
	
	//std::cout << iterG << " " << iterL << " " << epsL << " " << epsG << std::endl;
	while ((iterGlobal < iterG) && (resG>epsG)) {
#ifdef INSTRUMENTATION
		hipDeviceSynchronize();
		t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION

		updateLocalProbGPU(epsL2, iterL);
#ifdef INSTRUMENTATION
		hipDeviceSynchronize();
		t2 = std::chrono::high_resolution_clock::now();
		timePerBlock.increment(0, 1, (float) std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
#endif // INSTRUMENTATION

		tradeLin.swap(&Tlocal); // echange juste les pointeurs	
		updateGlobalProbGPU();
		if (!(iterGlobal % stepG)) {
#ifdef INSTRUMENTATION
			hipDeviceSynchronize();
			t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION

			resG = updateResEndo(iterGlobal / stepG);
#ifdef INSTRUMENTATION
			hipDeviceSynchronize();
			t2 = std::chrono::high_resolution_clock::now();
			timePerBlock.increment(0, 6, (float) std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
#endif // INSTRUMENTATION
		}
		//std::cout << iterGlobal << " " << iterLocal << " " << resL << " " << resF.get(0, iterGlobal / stepG) << " " << resF.get(1, iterGlobal / stepG) << std::endl;
		iterGlobal++;
	}
#ifdef INSTRUMENTATION
	occurencePerBlock.increment(0, 1, iterGlobal);
	occurencePerBlock.increment(0, 3, iterGlobal);
	occurencePerBlock.increment(0, 4, iterGlobal);
	occurencePerBlock.increment(0, 5, iterGlobal);
	occurencePerBlock.increment(0, 6, iterGlobal / stepG);

	hipDeviceSynchronize();
	t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION
	

	//std::cout << "fin simu temps : " << (float)(clock() - t) / CLOCKS_PER_SEC << std::endl;

	
	Kappa1.projectNeg(); //delta1
	Kappa2.projectNeg(); // delta2

	float fc = calcFc();
	//std::cout << iterGlobal << " " << iterLocal << " " << resL << " " << resG << std::endl;
	MatrixCPU tradeLinCPU;
	tradeLin.toMatCPU(tradeLinCPU);
	MatrixCPU LAMBDALinCPU;
	LAMBDALin.toMatCPU(LAMBDALinCPU);
	MatrixCPU PnCPU;
	Pn.toMatCPU(PnCPU);
	MatrixCPU MUCPU;
	MU.toMatCPU(MUCPU);
	MatrixCPU delta1CPU;
	Kappa1.toMatCPU(delta1CPU);
	MatrixCPU delta2CPU;
	Kappa2.toMatCPU(delta2CPU);
	int indice = 0;
	for (int idAgent = 0;idAgent < _nAgent; idAgent++) {
		MatrixCPU omega(cas.getVoisin(idAgent));
		int Nvoisinmax = nVoisinCPU.get(idAgent, 0);
		for (int voisin = 0; voisin < Nvoisinmax; voisin++) {
			int idVoisin = omega.get(voisin, 0);
			trade.set(idAgent, idVoisin, tradeLinCPU.get(indice, 0));
			LAMBDA.set(idAgent, idVoisin, LAMBDALinCPU.get(indice, 0));
			indice = indice + 1;
		}
	}
	if (cas.isAC()) {
		MatrixCPU tradeTot(2 * _nAgent, _nAgent);
		MatrixCPU LAMBDATot(2 * _nAgent, _nAgent);
		MatrixCPU PnTot(2 * _nAgent, 1);
		MatrixCPU MUTot(2 * _nAgent, 1);

		for (int n = 0; n < _nAgent; n++) {
			for (int m = 0; m < _nAgent; m++) {
				tradeTot.set(n, m, trade.get(n, m));
				LAMBDATot.set(n, m, LAMBDA.get(n, m));
			}
			PnTot.set(n, 0, PnCPU.get(n, 0));
			MUTot.set(n, 0, MUCPU.get(n, 0));
		}
		result->setLAMBDA(&LAMBDATot);
		result->setTrade(&tradeTot);
		result->setMU(&MUTot);
		result->setPn(&PnTot);

	}
	else {
		result->setLAMBDA(&LAMBDA);
		result->setTrade(&trade);
		result->setPn(&PnCPU);
		result->setMU(&MUCPU);

	}
	result->setResF(&resF);
	
	result->setDelta1(&delta1CPU);
	result->setDelta2(&delta2CPU);
	result->setIter(iterGlobal);
	
	result->setFc(fc);
	
	//result->setRho(_rhog);

#ifdef INSTRUMENTATION
	hipDeviceSynchronize();
	t2 = std::chrono::high_resolution_clock::now();
	timePerBlock.increment(0, 7, (float) std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
	occurencePerBlock.increment(0, 7, 1);
	result->setTimeBloc(&timePerBlock, &occurencePerBlock);
#endif // INSTRUMENTATION
	

	tall = clock() - tall;
	result->setTime((float)tall / CLOCKS_PER_SEC);
}

void ADMMGPUConst4::updateLocalProbGPU(float epsL, int nIterL) {
	int numBlocks = _nAgent;
	switch (_blockSize) {
	case 512:
		updateTradePGPUSharedResidual<512> << <numBlocks, _blockSize >> > (Tlocal._matrixGPU, Tlocal_pre._matrixGPU, Tmoy._matrixGPU, P._matrixGPU, MU._matrixGPU, nVoisin._matrixGPU, _at1, _at2, Bt1._matrixGPU, Ct._matrixGPU,
			matLb._matrixGPU, matUb._matrixGPU, Ap1._matrixGPU, Ap12._matrixGPU, Cp._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, CoresAgentLin._matrixGPU, epsL, nIterL);
		break;
	case 256:
		updateTradePGPUSharedResidual<256> << <numBlocks, _blockSize >> > (Tlocal._matrixGPU, Tlocal_pre._matrixGPU, Tmoy._matrixGPU, P._matrixGPU, MU._matrixGPU, nVoisin._matrixGPU, _at1, _at2, Bt1._matrixGPU, Ct._matrixGPU,
			matLb._matrixGPU, matUb._matrixGPU, Ap1._matrixGPU, Ap12._matrixGPU, Cp._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, CoresAgentLin._matrixGPU, epsL, nIterL);
		break;
	case 128:
		updateTradePGPUSharedResidual<128> << <numBlocks, _blockSize >> > (Tlocal._matrixGPU, Tlocal_pre._matrixGPU, Tmoy._matrixGPU, P._matrixGPU, MU._matrixGPU, nVoisin._matrixGPU, _at1, _at2, Bt1._matrixGPU, Ct._matrixGPU,
			matLb._matrixGPU, matUb._matrixGPU, Ap1._matrixGPU, Ap12._matrixGPU, Cp._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, CoresAgentLin._matrixGPU, epsL, nIterL);
		break;
	case 64:
		updateTradePGPUSharedResidual< 64> << <numBlocks, _blockSize >> > (Tlocal._matrixGPU, Tlocal_pre._matrixGPU, Tmoy._matrixGPU, P._matrixGPU, MU._matrixGPU, nVoisin._matrixGPU, _at1, _at2, Bt1._matrixGPU, Ct._matrixGPU,
			matLb._matrixGPU, matUb._matrixGPU, Ap1._matrixGPU, Ap12._matrixGPU, Cp._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, CoresAgentLin._matrixGPU, epsL, nIterL);
		break;
	case 32:
		updateTradePGPUSharedResidual< 32> << <numBlocks, _blockSize >> > (Tlocal._matrixGPU, Tlocal_pre._matrixGPU, Tmoy._matrixGPU, P._matrixGPU, MU._matrixGPU, nVoisin._matrixGPU, _at1, _at2, Bt1._matrixGPU, Ct._matrixGPU,
			matLb._matrixGPU, matUb._matrixGPU, Ap1._matrixGPU, Ap12._matrixGPU, Cp._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, CoresAgentLin._matrixGPU, epsL, nIterL);
		break;
	case 16:
		updateTradePGPUSharedResidual< 16> << <numBlocks, _blockSize >> > (Tlocal._matrixGPU, Tlocal_pre._matrixGPU, Tmoy._matrixGPU, P._matrixGPU, MU._matrixGPU, nVoisin._matrixGPU, _at1, _at2, Bt1._matrixGPU, Ct._matrixGPU,
			matLb._matrixGPU, matUb._matrixGPU, Ap1._matrixGPU, Ap12._matrixGPU, Cp._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, CoresAgentLin._matrixGPU, epsL, nIterL);
		break;
	case  8:
		updateTradePGPUSharedResidual<  8> << <numBlocks, _blockSize >> > (Tlocal._matrixGPU, Tlocal_pre._matrixGPU, Tmoy._matrixGPU, P._matrixGPU, MU._matrixGPU, nVoisin._matrixGPU, _at1, _at2, Bt1._matrixGPU, Ct._matrixGPU,
			matLb._matrixGPU, matUb._matrixGPU, Ap1._matrixGPU, Ap12._matrixGPU, Cp._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, CoresAgentLin._matrixGPU, epsL, nIterL);
		break;
	case  4:
		updateTradePGPUSharedResidual<  4> << <numBlocks, _blockSize >> > (Tlocal._matrixGPU, Tlocal_pre._matrixGPU, Tmoy._matrixGPU, P._matrixGPU, MU._matrixGPU, nVoisin._matrixGPU, _at1, _at2, Bt1._matrixGPU, Ct._matrixGPU,
			matLb._matrixGPU, matUb._matrixGPU, Ap1._matrixGPU, Ap12._matrixGPU, Cp._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, CoresAgentLin._matrixGPU, epsL, nIterL);
		break;
	case  2:
		updateTradePGPUSharedResidual<  2> << <numBlocks, _blockSize >> > (Tlocal._matrixGPU, Tlocal_pre._matrixGPU, Tmoy._matrixGPU, P._matrixGPU, MU._matrixGPU, nVoisin._matrixGPU, _at1, _at2, Bt1._matrixGPU, Ct._matrixGPU,
			matLb._matrixGPU, matUb._matrixGPU, Ap1._matrixGPU, Ap12._matrixGPU, Cp._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, CoresAgentLin._matrixGPU, epsL, nIterL);
		break;
	case  1:
		updateTradePGPUSharedResidual<  1> << <numBlocks, _blockSize >> > (Tlocal._matrixGPU, Tlocal_pre._matrixGPU, Tmoy._matrixGPU, P._matrixGPU, MU._matrixGPU, nVoisin._matrixGPU, _at1, _at2, Bt1._matrixGPU, Ct._matrixGPU,
			matLb._matrixGPU, matUb._matrixGPU, Ap1._matrixGPU, Ap12._matrixGPU, Cp._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, CoresAgentLin._matrixGPU, epsL, nIterL);
		break;
	}
}



void ADMMGPUConst4::updateGlobalProbGPU()
{
	//Rem : tout calcul qui est de taille N ou M peut �tre fait par les agents
		// Si le calcul est de taile L, soit c'est calcul� par un/des superviseurs, soit tous les agents le calcul (un peu absurde)

#ifdef INSTRUMENTATION
	// FB 3a
	hipDeviceSynchronize();
	std::chrono::high_resolution_clock::time_point t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION


	updatePnGPU << <_numBlocksN, _blockSize >> > (Pn._matrixGPU, Tmoy._matrixGPU, nVoisin._matrixGPU, _nAgent);
#ifdef INSTRUMENTATION
	hipDeviceSynchronize();
	std::chrono::high_resolution_clock::time_point t2 = std::chrono::high_resolution_clock::now();

	timePerBlock.increment(0, 3, (float) std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());

	// FB 3b
	hipDeviceSynchronize();
	t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION	

	updateAlphaTrans << < _numBlocksNL, _blockSize >> > (alpha._matrixGPU, GTrans._matrixGPU, Pn._matrixGPU, _nLine, _nAgent);
	updateQpartTrans << < _nLine, _blockSize, _nAgent * sizeof(float) >> > (Qpart._matrixGPU, alpha._matrixGPU, _nAgent, _nLine);
	updateQtotTrans << <_numBlocksL, _blockSize >> > (Qtot._matrixGPU, Qpart._matrixGPU, alpha._matrixGPU, _nLine);


#ifdef INSTRUMENTATION
	hipDeviceSynchronize();
	t2 = std::chrono::high_resolution_clock::now();
	timePerBlock.increment(0, 4, (float) std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());

	// FB 3c
	hipDeviceSynchronize();
	t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION
	Kappa1_pre.set(&Kappa1);
	Kappa2_pre.set(&Kappa2);
	updateKappaGPU << <_numBlocksL, _blockSize >> > (Kappa1._matrixGPU, Kappa2._matrixGPU, lLimit._matrixGPU, Qtot._matrixGPU, _nLine);
	diffKappa << <_numBlocksL, _blockSize >> > (tempL1._matrixGPU, Kappa1._matrixGPU, Kappa2._matrixGPU, _nLine);
	int numBlocks = _nAgent;
	switch (_blockSize) {
	case 512:
		updateCp2GPUTrans<512> << <numBlocks, _blockSize >> > (Cp2._matrixGPU, tempL1._matrixGPU, GTrans._matrixGPU, Qpart._matrixGPU, nVoisin._matrixGPU, _rho1, _nLine, _nAgent);
		break;
	case 256:
		updateCp2GPUTrans<256> << <numBlocks, _blockSize >> > (Cp2._matrixGPU, tempL1._matrixGPU, GTrans._matrixGPU, Qpart._matrixGPU, nVoisin._matrixGPU, _rho1, _nLine, _nAgent);
		break;
	case 128:
		updateCp2GPUTrans<128> << <numBlocks, _blockSize >> > (Cp2._matrixGPU, tempL1._matrixGPU, GTrans._matrixGPU, Qpart._matrixGPU, nVoisin._matrixGPU, _rho1, _nLine, _nAgent);
		break;
	case 64:
		updateCp2GPUTrans<64> << <numBlocks, _blockSize >> > (Cp2._matrixGPU, tempL1._matrixGPU, GTrans._matrixGPU, Qpart._matrixGPU, nVoisin._matrixGPU, _rho1, _nLine, _nAgent);
		break;
	case 32:
		updateCp2GPUTrans<32> << <numBlocks, _blockSize >> > (Cp2._matrixGPU, tempL1._matrixGPU, GTrans._matrixGPU, Qpart._matrixGPU, nVoisin._matrixGPU, _rho1, _nLine, _nAgent);
		break;
	case 16:
		updateCp2GPUTrans<16> << <numBlocks, _blockSize >> > (Cp2._matrixGPU, tempL1._matrixGPU, GTrans._matrixGPU, Qpart._matrixGPU, nVoisin._matrixGPU, _rho1, _nLine, _nAgent);
		break;
	case  8:
		updateCp2GPUTrans<8> << <numBlocks, _blockSize >> > (Cp2._matrixGPU, tempL1._matrixGPU, GTrans._matrixGPU, Qpart._matrixGPU, nVoisin._matrixGPU, _rho1, _nLine, _nAgent);
		break;
	case  4:
		updateCp2GPUTrans<4> << <numBlocks, _blockSize >> > (Cp2._matrixGPU, tempL1._matrixGPU, GTrans._matrixGPU, Qpart._matrixGPU, nVoisin._matrixGPU, _rho1, _nLine, _nAgent);
		break;
	case  2:
		updateCp2GPUTrans<2> << <numBlocks, _blockSize >> > (Cp2._matrixGPU, tempL1._matrixGPU, GTrans._matrixGPU, Qpart._matrixGPU, nVoisin._matrixGPU, _rho1, _nLine, _nAgent);
		break;
	case  1:
		updateCp2GPUTrans<1> << <numBlocks, _blockSize >> > (Cp2._matrixGPU, tempL1._matrixGPU, GTrans._matrixGPU, Qpart._matrixGPU, nVoisin._matrixGPU, _rho1, _nLine, _nAgent);
		break;
	}

	updateLAMBDABt1GPU << <_numBlocksM, _blockSize >> > (Bt1._matrixGPU, LAMBDALin._matrixGPU, tradeLin._matrixGPU, _rhog, CoresLinTrans._matrixGPU, _nTrade);
	updateCp << <_numBlocksN, _blockSize >> > (Cp._matrixGPU, Cp1._matrixGPU, Cp2._matrixGPU, _nAgent);

#ifdef INSTRUMENTATION
	hipDeviceSynchronize();
	t2 = std::chrono::high_resolution_clock::now();
	timePerBlock.increment(0, 5, (float) std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
#endif // INSTRUMENTATION
	

}



