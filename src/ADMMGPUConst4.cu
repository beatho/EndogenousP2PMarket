#include "hip/hip_runtime.h"
#include "../head/ADMMGPUConst4.cuh"

ADMMGPUConst4::ADMMGPUConst4() : MethodP2P()
{
#if DEBUG_CONSTRUCTOR
	std::cout << "Constructeur ADMMGPUConst4" << std::endl;
#endif // DEBUG_CONSTRUCTOR
	_name = NAME;
}


ADMMGPUConst4::ADMMGPUConst4(float rho) : MethodP2P()
{
#if DEBUG_CONSTRUCTOR
	std::cout << "Constructeur ADMMGPUConst4 defaut" << std::endl;
#endif // DEBUG_CONSTRUCTOR
	_name = NAME;
	_rho = rho;
}

ADMMGPUConst4::~ADMMGPUConst4()
{
}

void ADMMGPUConst4::setParam(float rho)
{
	_rho = rho;
}

void ADMMGPUConst4::setTau(float tau)
{
	if (tau < 1) {
		throw std::invalid_argument("tau must be greater than 1");
	}
	_tau = tau;
}

void ADMMGPUConst4::init(const Simparam& sim, const StudyCase& cas)
{
	// intitilisation des matrixs et variables 
	clock_t t = clock();
	_rhog = sim.getRho();
	_rho1 = sim.getRho1();
	//std::cout << "rho initial " << _rhog << std::endl;
	_nAgent = sim.getNAgent();
	
	_rhol = _rho;
	if (_rho == 0) {
		_rhol = _rhog;
	}
	const int iterG = sim.getIterG();
	const int stepG = sim.getStepG();
	float epsG = sim.getEpsG();
	float epsGC = sim.getEpsGC();
	_ratioEps = epsG / epsGC;
	
	if (cas.isAC()) {
		MatrixCPU nVoisinT = cas.getNvoi();
		nVoisinCPU = MatrixCPU(_nAgent, 1);
		for (int n = 0; n < _nAgent; n++) {
			nVoisinCPU.set(n, 0, nVoisinT.get(n, 0));
		}
	}
	else {
		nVoisinCPU = cas.getNvoi();
	}
	
	nVoisin = MatrixGPU(nVoisinCPU, 1);
	nVoisin.preallocateReduction();
	
	int nVoisinMax = nVoisin.max2();
	if (_blockSize * NMAXPEERPERTRHREAD < nVoisinMax) {
		std::cout << _blockSize << " " << NMAXPEERPERTRHREAD << " " << nVoisinMax << std::endl;
		throw std::invalid_argument("For this Method, an agent must not have more than 5120 peers");
	}

	_nLine = cas.getNLine();
	
	_nBus = cas.getNBus();

	_nTrade = nVoisin.sum();
	//std::cout << "nVoisinma " << nVoisinMax << "trade " << _nTrade << std::endl;
	_numBlocksN = ceil((_nAgent + _blockSize - 1) / _blockSize);
	_numBlocksM = ceil((_nTrade + _blockSize - 1) / _blockSize);
	_numBlocksL = ceil((_nLine + _blockSize - 1) / _blockSize);
	_numBlocksNL = ceil((_nAgent * _nLine + _blockSize - 1) / _blockSize);
	_at1 = _rhog; // represente en fait 2*a
	_at2 = _rhol;

	resF = MatrixCPU(3, (iterG / stepG) + 1);
	resX = MatrixCPU(4, (iterG / stepG) + 1);

	MatrixCPU BETA(cas.getBeta());
	MatrixGPU Ub(cas.getUb());
	MatrixGPU Lb(cas.getLb());
	LAMBDA = sim.getLambda();
	trade = sim.getTrade();
	
	//std::cout << "mise sous forme lin�aire" << std::endl;
	// Rem : si matrice d�j� existante, elles sont d�j� sur GPU donc bug pour les get
	if (Ct.getPos()) { // une copie en trop mais pour l'instant c'est ok...
		CoresMatLin.transferCPU();

		CoresLinAgent.transferCPU();
		CoresAgentLin.transferCPU();
		CoresLinVoisin.transferCPU();
		CoresLinTrans.transferCPU();

		Tlocal_pre.transferCPU();
		tradeLin.transferCPU();
		LAMBDALin.transferCPU();

		matLb.transferCPU();
		matUb.transferCPU();
		Ct.transferCPU();

		if (cas.isAC()) {
			a.transferCPU();
			b.transferCPU();
			Pmin.transferCPU();
			Pmax.transferCPU();
			MU.transferCPU();
			Tmoy.transferCPU();
		}
	}


	CoresMatLin = MatrixGPU(_nAgent, _nAgent, -1);
	CoresLinAgent = MatrixGPU(_nTrade, 1);
	CoresAgentLin = MatrixGPU(_nAgent + 1, 1);
	CoresLinVoisin = MatrixGPU(_nTrade, 1);
	CoresLinTrans = MatrixGPU(_nTrade, 1);

	Tlocal_pre = MatrixGPU(_nTrade, 1);
	tradeLin = MatrixGPU(_nTrade, 1);
	LAMBDALin = MatrixGPU(_nTrade, 1);

	matLb = MatrixGPU(_nTrade, 1);
	matUb = MatrixGPU(_nTrade, 1);
	Ct = MatrixGPU(_nTrade, 1);

	int indice = 0;

	for (int idAgent = 0; idAgent < _nAgent; idAgent++) {
		MatrixCPU omega(cas.getVoisin(idAgent));
		int Nvoisinmax = nVoisinCPU.get(idAgent, 0);
		for (int voisin = 0; voisin < Nvoisinmax; voisin++) {
			int idVoisin = omega.get(voisin, 0);
			matLb.set(indice, 0, Lb.get(idAgent, 0));
			matUb.set(indice, 0, Ub.get(idAgent, 0));
			Ct.set(indice, 0, BETA.get(idAgent, idVoisin));
			tradeLin.set(indice, 0, trade.get(idAgent, idVoisin));
			Tlocal_pre.set(indice, 0, trade.get(idAgent, idVoisin));
			LAMBDALin.set(indice, 0, LAMBDA.get(idAgent, idVoisin));
			CoresLinAgent.set(indice, 0, idAgent);
			CoresLinVoisin.set(indice, 0, idVoisin);
			CoresMatLin.set(idAgent, idVoisin, indice);
			indice = indice + 1;
		}
		CoresAgentLin.set(idAgent + 1, 0, indice);
	}
	for (int lin = 0; lin < _nTrade; lin++) {
		int i = CoresLinAgent.get(lin, 0);
		int j = CoresLinVoisin.get(lin, 0);
		int k = CoresMatLin.get(j, i);
		CoresLinTrans.set(lin, 0, k);
	}


	// transfert des mises lineaire
	matUb.transferGPU();
	matLb.transferGPU();
	Ct.transferGPU();

	Tlocal_pre.transferGPU();
	tradeLin.transferGPU();
	LAMBDALin.transferGPU();

	CoresAgentLin.transferGPU();
	CoresLinAgent.transferGPU();
	CoresLinVoisin.transferGPU();
	CoresMatLin.transferGPU();
	CoresLinTrans.transferGPU();

	//std::cout << "donnees sur GPU pour le grid" << std::endl;
	Kappa1 = MatrixGPU(_nLine, 1, 0, 1);
	Kappa2 = MatrixGPU(_nLine, 1, 0, 1);
	Kappa1_pre = MatrixGPU(_nLine, 1, 0, 1);
	Kappa2_pre = MatrixGPU(_nLine, 1, 0, 1);
	Qpart = MatrixGPU(_nAgent, _nLine, 0, 1);
	Qtot = MatrixGPU(_nLine, 1, 0, 1);
	alpha = MatrixGPU(_nAgent, _nLine, 0, 1);
	G = MatrixGPU(cas.getPowerSensi());

	lLimit = MatrixGPU(cas.getLineLimit(), 1);

	GTrans = MatrixGPU(_nAgent, _nLine);

	if (GTrans.getPos()) {
		GTrans.transferCPU();
	}

	GTrans.setTrans(&G);
	//G.transferGPU();

	GTrans.transferGPU();

	G2 = GTrans;

	G2.multiplyT(&GTrans);

	//std::cout << "autres donn�e sur GPU" << std::endl;
	tempNN = MatrixGPU(_nTrade, 1, 0, 1);
	tempN1 = MatrixGPU(_nAgent, 1, 0, 1); // plut�t que de re-allouer de la m�moire � chaque utilisation
	tempL1 = MatrixGPU(_nLine, 1, 0, 1);
	tempL2 = MatrixGPU(_nLine, 1, 0, 1);
	//MatrixGPU temp1N(1, _nAgent, 0, 1);

	Tlocal = MatrixGPU(_nTrade, 1, 0, 1);
	P = MatrixGPU(_nAgent, 1, 0, 1); // moyenne des trades
	Pn = MatrixGPU(_nAgent, 1, 0, 1); // somme des trades


	if (cas.isAC()) {
		MatrixCPU aT = cas.geta();
		MatrixCPU bT = cas.getb();
		MatrixCPU PminT = cas.getPmin();
		MatrixCPU PmaxT = cas.getPmax();
		MatrixCPU MUT = sim.getMU(); // facteur reduit i.e lambda_l/_rho
		MatrixCPU TmoyT = sim.getPn();
		a = MatrixGPU(_nAgent, 1);
		b = MatrixGPU(_nAgent, 1);
		Pmin = MatrixGPU(_nAgent, 1);
		Pmax = MatrixGPU(_nAgent, 1);
		MU = MatrixGPU(_nAgent, 1);
		Tmoy = MatrixGPU(_nAgent, 1);

		for (int n = 0; n < _nAgent; n++) {
			a.set(n, 0, aT.get(n, 0));
			b.set(n, 0, bT.get(n, 0));
			Pmin.set(n, 0, PminT.get(n, 0));
			Pmax.set(n, 0, PmaxT.get(n, 0));
			MU.set(n, 0, MUT.get(n, 0));
			Tmoy.set(n, 0, TmoyT.get(n, 0));
		}
		a.transferGPU();
		b.transferGPU();
		Pmin.transferGPU();
		Pmax.transferGPU();
		MU.transferGPU();
		Tmoy.transferGPU();
	}
	else {
		a = MatrixGPU(cas.geta(), 1);
		b = MatrixGPU(cas.getb(), 1);

		Pmin = MatrixGPU(cas.getPmin(), 1);
		Pmax = MatrixGPU(cas.getPmax(), 1);
		MU = MatrixGPU(sim.getMU(), 1); // facteur reduit i.e lambda_l/_rho
		Tmoy = MatrixGPU(sim.getPn(), 1);
	}
	Ap1 = nVoisin;
	Ap2 = a;
	Cp1 = b;

	Ap12 = MatrixGPU(_nAgent, 1, 0, 1);
	Ap2a = a;
	Ap2b = MatrixGPU(_nAgent, 1, 0, 1);

	Bt1 = MatrixGPU(_nTrade, 1, 0, 1);
	Cp = MatrixGPU(_nAgent, 1, 0, 1);
	Cp2 = MatrixGPU(_nAgent, 1, 0, 1);
	Cp1 = b;

	

	CHECK_LAST_CUDA_ERROR();
	tempNN.preallocateReduction();
	Tlocal.preallocateReduction();
	tempL1.preallocateReduction();

	P.preallocateReduction();

	Pmin.divideT(&nVoisin);
	Pmax.divideT(&nVoisin);
	Ap1.multiply(_rhol);
	Cp1.multiplyT(&nVoisin);
	Tmoy.divideT(&nVoisin);


	tempN1.sum(&G2);
	tempN1.multiply(2 * _rho1);
	Ap2.add(&tempN1);
	Ap2.multiplyT(&nVoisin);
	Ap2.multiplyT(&nVoisin);
	Ap12.add(&Ap1, &Ap2);
	
	/*Ap2b.sum(&G2);
	Ap2b.multiply(2 * _rho1);
	Ap2a.multiplyT(&nVoisin);
	Ap2a.multiplyT(&nVoisin);
	Ap2b.multiplyT(&nVoisin);
	Ap2b.multiplyT(&nVoisin);

	Ap2.add(&Ap2a, &Ap2b);*/

	updateGlobalProbGPU();
	//std::cout << "rho " << _rhog << " rhoL " << _rhol << " _rho1 " << _rho1 << std::endl;
	//std::cout << "fin init temps : " << (float)(clock() - t) / CLOCKS_PER_SEC << std::endl;
	
}



void ADMMGPUConst4::updateP0(const StudyCase& cas)
{
	_id = _id + 1;
#ifdef INSTRUMENTATION
	hipDeviceSynchronize();
	std::chrono::high_resolution_clock::time_point t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION
	

	matLb.transferCPU();
	
	b.transferCPU();
	Pmin.transferCPU();
	Pmax.transferCPU();
	if (cas.isAC()) {
		
		MatrixCPU bT = cas.getb();
		MatrixCPU PminT = cas.getPmin();
		MatrixCPU PmaxT = cas.getPmax();


		for (int n = 0; n < _nAgent; n++) {
			
			b.set(n, 0, bT.get(n, 0));
			Pmin.set(n, 0, PminT.get(n, 0));
			Pmax.set(n, 0, PmaxT.get(n, 0));
		}
		
		b.transferGPU();
		Pmin.transferGPU();
		Pmax.transferGPU();
	}
	else {
	
		b = cas.getb();
		Pmin = cas.getPmin();
		Pmax = cas.getPmax();
		
	}


	


	MatrixGPU Lb(cas.getLb());

	Cp1 = b;
	int indice = 0;

	for (int idAgent = 0; idAgent < _nAgent; idAgent++) {
		int Nvoisinmax = nVoisinCPU.get(idAgent, 0);
		for (int voisin = 0; voisin < Nvoisinmax; voisin++) {
			matLb.set(indice, 0, Lb.get(idAgent, 0));
			indice = indice + 1;
		}
	}

	
	matLb.transferGPU();

	Pmin.divideT(&nVoisin);
	Pmax.divideT(&nVoisin);
	Cp1.multiplyT(&nVoisin);
#ifdef INSTRUMENTATION
	hipDeviceSynchronize();
	std::chrono::high_resolution_clock::time_point t2 = std::chrono::high_resolution_clock::now();
	timePerBlock.increment(0, 8, std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
	occurencePerBlock.increment(0, 8, 1);
#endif // INSTRUMENTATION

}


void ADMMGPUConst4::solve(Simparam* result, const Simparam& sim, const StudyCase& cas)
{
#ifdef DEBUG_SOLVE
	cas.display();
	sim.display(1);
#endif // DEBUG_SOLVE
	
	
	clock_t tall = clock();

#ifdef INSTRUMENTATION
	std::chrono::high_resolution_clock::time_point t1;
	std::chrono::high_resolution_clock::time_point t2;	
#endif // INSTRUMENTATION




	if (_id == 0) {
#ifdef INSTRUMENTATION
		hipDeviceSynchronize();
		t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION
		init(sim, cas);
#ifdef INSTRUMENTATION
		hipDeviceSynchronize();
		t2 = std::chrono::high_resolution_clock::now();
		timePerBlock.increment(0, 0, std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
		occurencePerBlock.increment(0, 0, 1);
#endif // INSTRUMENTATION
	}
	_rhog = sim.getRho();
	//std::cout << _numBlocks2 << " " <<  _blockSize << std::endl;
	/*_mu = _muInit;
	_mu1 = _muInit;
	_rhog = sim.getRho();
	float oldRho1 = _rho1;
	_rho1 = sim.getRho1();
	float tau = _rho1 / oldRho1;
	Ap2b.multiply(_tau);
	Ap2.add(&Ap2a, &Ap2b);
	Ap12.add(&Ap1, &Ap2);*/

	_at1 = _rhog; // represente en fait 2*a
	
	float epsG = sim.getEpsG();
	float epsL = sim.getEpsL();
	const int stepL = sim.getStepL();
	const int stepG = sim.getStepG();
	const int iterG = sim.getIterG();
	const int iterL = sim.getIterL();
	

	float resG = 2 * epsG;
	float epsL2 = epsL * epsL;
	int iterGlobal = 0;
	int iterLocal = 0;
	
	//std::cout << iterG << " " << iterL << " " << epsL << " " << epsG << std::endl;
	while ((iterGlobal < iterG) && (resG>epsG)) {
#ifdef INSTRUMENTATION
		hipDeviceSynchronize();
		t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION

		updateLocalProbGPU(epsL2, iterL);
#ifdef INSTRUMENTATION
		hipDeviceSynchronize();
		t2 = std::chrono::high_resolution_clock::now();
		timePerBlock.increment(0, 1, std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
#endif // INSTRUMENTATION

		tradeLin.swap(&Tlocal); // echange juste les pointeurs	
		updateGlobalProbGPU();
		if (!(iterGlobal % stepG)) {
#ifdef INSTRUMENTATION
			hipDeviceSynchronize();
			t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION

			resG = updateRes(&resF, &Tlocal, iterGlobal / stepG, &tempNN);
#ifdef INSTRUMENTATION
			hipDeviceSynchronize();
			t2 = std::chrono::high_resolution_clock::now();
			timePerBlock.increment(0, 6, std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
#endif // INSTRUMENTATION
		}
		//std::cout << iterGlobal << " " << iterLocal << " " << resL << " " << resF.get(0, iterGlobal / stepG) << " " << resF.get(1, iterGlobal / stepG) << std::endl;
		iterGlobal++;
	}
#ifdef INSTRUMENTATION
	occurencePerBlock.increment(0, 1, iterGlobal);
	occurencePerBlock.increment(0, 3, iterGlobal);
	occurencePerBlock.increment(0, 4, iterGlobal);
	occurencePerBlock.increment(0, 5, iterGlobal);
	occurencePerBlock.increment(0, 6, iterGlobal / stepG);

	hipDeviceSynchronize();
	t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION
	

	//std::cout << "fin simu temps : " << (float)(clock() - t) / CLOCKS_PER_SEC << std::endl;

	
	Kappa1.projectNeg(); //delta1
	Kappa2.projectNeg(); // delta2

	float fc = calcFc(&a, &b, &tradeLin, &Pn, &Ct, &tempN1, &tempNN);
	//std::cout << iterGlobal << " " << iterLocal << " " << resL << " " << resG << std::endl;
	MatrixCPU tradeLinCPU;
	tradeLin.toMatCPU(tradeLinCPU);
	MatrixCPU LAMBDALinCPU;
	LAMBDALin.toMatCPU(LAMBDALinCPU);
	MatrixCPU PnCPU;
	Pn.toMatCPU(PnCPU);
	MatrixCPU MUCPU;
	MU.toMatCPU(MUCPU);
	MatrixCPU delta1CPU;
	Kappa1.toMatCPU(delta1CPU);
	MatrixCPU delta2CPU;
	Kappa2.toMatCPU(delta2CPU);
	int indice = 0;
	for (int idAgent = 0;idAgent < _nAgent; idAgent++) {
		MatrixCPU omega(cas.getVoisin(idAgent));
		int Nvoisinmax = nVoisinCPU.get(idAgent, 0);
		for (int voisin = 0; voisin < Nvoisinmax; voisin++) {
			int idVoisin = omega.get(voisin, 0);
			trade.set(idAgent, idVoisin, tradeLinCPU.get(indice, 0));
			LAMBDA.set(idAgent, idVoisin, LAMBDALinCPU.get(indice, 0));
			indice = indice + 1;
		}
	}
	if (cas.isAC()) {
		MatrixCPU tradeTot(2 * _nAgent, _nAgent);
		MatrixCPU LAMBDATot(2 * _nAgent, _nAgent);
		MatrixCPU PnTot(2 * _nAgent, 1);
		MatrixCPU MUTot(2 * _nAgent, 1);

		for (int n = 0; n < _nAgent; n++) {
			for (int m = 0; m < _nAgent; m++) {
				tradeTot.set(n, m, trade.get(n, m));
				LAMBDATot.set(n, m, LAMBDA.get(n, m));
			}
			PnTot.set(n, 0, PnCPU.get(n, 0));
			MUTot.set(n, 0, MUCPU.get(n, 0));
		}
		result->setLAMBDA(&LAMBDATot);
		result->setTrade(&tradeTot);
		result->setMU(&MUTot);
		result->setPn(&PnTot);

	}
	else {
		result->setLAMBDA(&LAMBDA);
		result->setTrade(&trade);
		result->setPn(&PnCPU);
		result->setMU(&MUCPU);

	}
	result->setResF(&resF);
	
	result->setDelta1(&delta1CPU);
	result->setDelta2(&delta2CPU);
	result->setIter(iterGlobal);
	
	result->setFc(fc);
	
	//result->setRho(_rhog);

#ifdef INSTRUMENTATION
	hipDeviceSynchronize();
	t2 = std::chrono::high_resolution_clock::now();
	timePerBlock.increment(0, 7, std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
	occurencePerBlock.increment(0, 7, 1);
	result->setTimeBloc(&timePerBlock, &occurencePerBlock);
#endif // INSTRUMENTATION
	

	tall = clock() - tall;
	result->setTime((float)tall / CLOCKS_PER_SEC);
}

void ADMMGPUConst4::updateLocalProbGPU(float epsL, int nIterL) {
	int numBlocks = _nAgent;
	switch (_blockSize) {
	case 512:
		updateTradePGPUSharedResidual<512> << <numBlocks, _blockSize >> > (Tlocal._matrixGPU, Tlocal_pre._matrixGPU, Tmoy._matrixGPU, P._matrixGPU, MU._matrixGPU, nVoisin._matrixGPU, _at1, _at2, Bt1._matrixGPU, Ct._matrixGPU,
			matLb._matrixGPU, matUb._matrixGPU, Ap1._matrixGPU, Ap12._matrixGPU, Cp._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, CoresAgentLin._matrixGPU, epsL, nIterL);
		break;
	case 256:
		updateTradePGPUSharedResidual<256> << <numBlocks, _blockSize >> > (Tlocal._matrixGPU, Tlocal_pre._matrixGPU, Tmoy._matrixGPU, P._matrixGPU, MU._matrixGPU, nVoisin._matrixGPU, _at1, _at2, Bt1._matrixGPU, Ct._matrixGPU,
			matLb._matrixGPU, matUb._matrixGPU, Ap1._matrixGPU, Ap12._matrixGPU, Cp._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, CoresAgentLin._matrixGPU, epsL, nIterL);
		break;
	case 128:
		updateTradePGPUSharedResidual<128> << <numBlocks, _blockSize >> > (Tlocal._matrixGPU, Tlocal_pre._matrixGPU, Tmoy._matrixGPU, P._matrixGPU, MU._matrixGPU, nVoisin._matrixGPU, _at1, _at2, Bt1._matrixGPU, Ct._matrixGPU,
			matLb._matrixGPU, matUb._matrixGPU, Ap1._matrixGPU, Ap12._matrixGPU, Cp._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, CoresAgentLin._matrixGPU, epsL, nIterL);
		break;
	case 64:
		updateTradePGPUSharedResidual< 64> << <numBlocks, _blockSize >> > (Tlocal._matrixGPU, Tlocal_pre._matrixGPU, Tmoy._matrixGPU, P._matrixGPU, MU._matrixGPU, nVoisin._matrixGPU, _at1, _at2, Bt1._matrixGPU, Ct._matrixGPU,
			matLb._matrixGPU, matUb._matrixGPU, Ap1._matrixGPU, Ap12._matrixGPU, Cp._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, CoresAgentLin._matrixGPU, epsL, nIterL);
		break;
	case 32:
		updateTradePGPUSharedResidual< 32> << <numBlocks, _blockSize >> > (Tlocal._matrixGPU, Tlocal_pre._matrixGPU, Tmoy._matrixGPU, P._matrixGPU, MU._matrixGPU, nVoisin._matrixGPU, _at1, _at2, Bt1._matrixGPU, Ct._matrixGPU,
			matLb._matrixGPU, matUb._matrixGPU, Ap1._matrixGPU, Ap12._matrixGPU, Cp._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, CoresAgentLin._matrixGPU, epsL, nIterL);
		break;
	case 16:
		updateTradePGPUSharedResidual< 16> << <numBlocks, _blockSize >> > (Tlocal._matrixGPU, Tlocal_pre._matrixGPU, Tmoy._matrixGPU, P._matrixGPU, MU._matrixGPU, nVoisin._matrixGPU, _at1, _at2, Bt1._matrixGPU, Ct._matrixGPU,
			matLb._matrixGPU, matUb._matrixGPU, Ap1._matrixGPU, Ap12._matrixGPU, Cp._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, CoresAgentLin._matrixGPU, epsL, nIterL);
		break;
	case  8:
		updateTradePGPUSharedResidual<  8> << <numBlocks, _blockSize >> > (Tlocal._matrixGPU, Tlocal_pre._matrixGPU, Tmoy._matrixGPU, P._matrixGPU, MU._matrixGPU, nVoisin._matrixGPU, _at1, _at2, Bt1._matrixGPU, Ct._matrixGPU,
			matLb._matrixGPU, matUb._matrixGPU, Ap1._matrixGPU, Ap12._matrixGPU, Cp._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, CoresAgentLin._matrixGPU, epsL, nIterL);
		break;
	case  4:
		updateTradePGPUSharedResidual<  4> << <numBlocks, _blockSize >> > (Tlocal._matrixGPU, Tlocal_pre._matrixGPU, Tmoy._matrixGPU, P._matrixGPU, MU._matrixGPU, nVoisin._matrixGPU, _at1, _at2, Bt1._matrixGPU, Ct._matrixGPU,
			matLb._matrixGPU, matUb._matrixGPU, Ap1._matrixGPU, Ap12._matrixGPU, Cp._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, CoresAgentLin._matrixGPU, epsL, nIterL);
		break;
	case  2:
		updateTradePGPUSharedResidual<  2> << <numBlocks, _blockSize >> > (Tlocal._matrixGPU, Tlocal_pre._matrixGPU, Tmoy._matrixGPU, P._matrixGPU, MU._matrixGPU, nVoisin._matrixGPU, _at1, _at2, Bt1._matrixGPU, Ct._matrixGPU,
			matLb._matrixGPU, matUb._matrixGPU, Ap1._matrixGPU, Ap12._matrixGPU, Cp._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, CoresAgentLin._matrixGPU, epsL, nIterL);
		break;
	case  1:
		updateTradePGPUSharedResidual<  1> << <numBlocks, _blockSize >> > (Tlocal._matrixGPU, Tlocal_pre._matrixGPU, Tmoy._matrixGPU, P._matrixGPU, MU._matrixGPU, nVoisin._matrixGPU, _at1, _at2, Bt1._matrixGPU, Ct._matrixGPU,
			matLb._matrixGPU, matUb._matrixGPU, Ap1._matrixGPU, Ap12._matrixGPU, Cp._matrixGPU, Pmin._matrixGPU, Pmax._matrixGPU, CoresAgentLin._matrixGPU, epsL, nIterL);
		break;
	}
}



void ADMMGPUConst4::updateGlobalProbGPU()
{
	//Rem : tout calcul qui est de taille N ou M peut �tre fait par les agents
		// Si le calcul est de taile L, soit c'est calcul� par un/des superviseurs, soit tous les agents le calcul (un peu absurde)

#ifdef INSTRUMENTATION
	// FB 3a
	hipDeviceSynchronize();
	std::chrono::high_resolution_clock::time_point t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION


	updatePnGPU << <_numBlocksN, _blockSize >> > (Pn._matrixGPU, Tmoy._matrixGPU, nVoisin._matrixGPU, _nAgent);
#ifdef INSTRUMENTATION
	hipDeviceSynchronize();
	std::chrono::high_resolution_clock::time_point t2 = std::chrono::high_resolution_clock::now();

	timePerBlock.increment(0, 3, std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());

	// FB 3b
	hipDeviceSynchronize();
	t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION	

	updateAlphaTrans << < _numBlocksNL, _blockSize >> > (alpha._matrixGPU, GTrans._matrixGPU, Pn._matrixGPU, _nLine, _nAgent);
	updateQpartTrans << < _nLine, _blockSize, _nAgent * sizeof(float) >> > (Qpart._matrixGPU, alpha._matrixGPU, _nAgent, _nLine);
	updateQtotTrans << <_numBlocksL, _blockSize >> > (Qtot._matrixGPU, Qpart._matrixGPU, alpha._matrixGPU, _nLine);


#ifdef INSTRUMENTATION
	hipDeviceSynchronize();
	t2 = std::chrono::high_resolution_clock::now();
	timePerBlock.increment(0, 4, std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());

	// FB 3c
	hipDeviceSynchronize();
	t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION
	Kappa1_pre.set(&Kappa1);
	Kappa2_pre.set(&Kappa2);
	updateKappaGPU << <_numBlocksL, _blockSize >> > (Kappa1._matrixGPU, Kappa2._matrixGPU, lLimit._matrixGPU, Qtot._matrixGPU, _nLine);
	diffKappa << <_numBlocksL, _blockSize >> > (tempL1._matrixGPU, Kappa1._matrixGPU, Kappa2._matrixGPU, _nLine);
	int numBlocks = _nAgent;
	switch (_blockSize) {
	case 512:
		updateCp2GPUTrans<512> << <numBlocks, _blockSize >> > (Cp2._matrixGPU, tempL1._matrixGPU, GTrans._matrixGPU, Qpart._matrixGPU, nVoisin._matrixGPU, _rho1, _nLine, _nAgent);
		break;
	case 256:
		updateCp2GPUTrans<256> << <numBlocks, _blockSize >> > (Cp2._matrixGPU, tempL1._matrixGPU, GTrans._matrixGPU, Qpart._matrixGPU, nVoisin._matrixGPU, _rho1, _nLine, _nAgent);
		break;
	case 128:
		updateCp2GPUTrans<128> << <numBlocks, _blockSize >> > (Cp2._matrixGPU, tempL1._matrixGPU, GTrans._matrixGPU, Qpart._matrixGPU, nVoisin._matrixGPU, _rho1, _nLine, _nAgent);
		break;
	case 64:
		updateCp2GPUTrans<64> << <numBlocks, _blockSize >> > (Cp2._matrixGPU, tempL1._matrixGPU, GTrans._matrixGPU, Qpart._matrixGPU, nVoisin._matrixGPU, _rho1, _nLine, _nAgent);
		break;
	case 32:
		updateCp2GPUTrans<32> << <numBlocks, _blockSize >> > (Cp2._matrixGPU, tempL1._matrixGPU, GTrans._matrixGPU, Qpart._matrixGPU, nVoisin._matrixGPU, _rho1, _nLine, _nAgent);
		break;
	case 16:
		updateCp2GPUTrans<16> << <numBlocks, _blockSize >> > (Cp2._matrixGPU, tempL1._matrixGPU, GTrans._matrixGPU, Qpart._matrixGPU, nVoisin._matrixGPU, _rho1, _nLine, _nAgent);
		break;
	case  8:
		updateCp2GPUTrans<8> << <numBlocks, _blockSize >> > (Cp2._matrixGPU, tempL1._matrixGPU, GTrans._matrixGPU, Qpart._matrixGPU, nVoisin._matrixGPU, _rho1, _nLine, _nAgent);
		break;
	case  4:
		updateCp2GPUTrans<4> << <numBlocks, _blockSize >> > (Cp2._matrixGPU, tempL1._matrixGPU, GTrans._matrixGPU, Qpart._matrixGPU, nVoisin._matrixGPU, _rho1, _nLine, _nAgent);
		break;
	case  2:
		updateCp2GPUTrans<2> << <numBlocks, _blockSize >> > (Cp2._matrixGPU, tempL1._matrixGPU, GTrans._matrixGPU, Qpart._matrixGPU, nVoisin._matrixGPU, _rho1, _nLine, _nAgent);
		break;
	case  1:
		updateCp2GPUTrans<1> << <numBlocks, _blockSize >> > (Cp2._matrixGPU, tempL1._matrixGPU, GTrans._matrixGPU, Qpart._matrixGPU, nVoisin._matrixGPU, _rho1, _nLine, _nAgent);
		break;
	}

	updateLAMBDABt1GPU << <_numBlocksM, _blockSize >> > (Bt1._matrixGPU, LAMBDALin._matrixGPU, tradeLin._matrixGPU, _rhog, CoresLinTrans._matrixGPU, _nTrade);
	updateCp << <_numBlocksN, _blockSize >> > (Cp._matrixGPU, Cp1._matrixGPU, Cp2._matrixGPU, _nAgent);

#ifdef INSTRUMENTATION
	hipDeviceSynchronize();
	t2 = std::chrono::high_resolution_clock::now();
	timePerBlock.increment(0, 5, std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
#endif // INSTRUMENTATION
	

}



float ADMMGPUConst4::updateRes(MatrixCPU* res, MatrixGPU* Tlocal, int iter, MatrixGPU* tempNN)
{
	float resS = Tlocal->max2(&tradeLin);

	updateDiffGPU <<<_numBlocksM, _blockSize >>> (tempNN->_matrixGPU, Tlocal->_matrixGPU, CoresLinTrans._matrixGPU, _nTrade);
	float resR = tempNN->max2();
	// version de l'article
	/*tempL1.set(&Kappa1);
	tempL2.set(&Kappa2);
	Kappa1_pre.projectNeg();
	Kappa2_pre.projectNeg();
	tempL1.projectNeg();
	tempL2.projectNeg();
	tempL1.subtract(&Kappa1_pre);
	tempL2.subtract(&Kappa2_pre);
	tempL1.multiplyT(&tempL1);
	tempL2.multiplyT(&tempL2);
	tempL1.add(&tempL2);*/
	updateResX << <_numBlocksL, _blockSize >> > (tempL1._matrixGPU, Kappa1._matrixGPU, Kappa2._matrixGPU, Kappa1_pre._matrixGPU, Kappa2_pre._matrixGPU, _nLine);

	float resXf = _ratioEps * sqrt(tempL1.max2());/**/

	// ma version se basant sur la th�orie pour 1 et 2
	// r = x-z = l-Qtot -max(0,Kappa1) 
	// s = -rho (z^k-z^{k-1}) = max(0,Kappa1^k) - max(0,Kappa1^{k-1})
	/*MatrixGPU KappaPos(Kappa1);
	MatrixGPU KappaPos2(Kappa2);
	MatrixGPU tempL(Kappa1_pre);
	MatrixGPU tempL2(Kappa2_pre);
	KappaPos.projectPos();
	KappaPos2.projectPos();
	tempL.projectPos();
	tempL2.projectPos();
	tempL.subtract(&KappaPos);
	tempL2.subtract(&KappaPos2);
	tempL.multiplyT(&tempL);
	tempL2.multiplyT(&tempL2);

	float s1 = sqrt(tempL.max2()); // aie aie aie si rho est grand !!!!!!!!!!!!!!!
	float s2 = sqrt(tempL2.max2()); // manque le *rho, est ce grave ?
	float resXf = MAX(s1,s2);

	tempL1.set(&lLimit);
	tempL1.subtract(&Qtot);
	tempL1.subtract(&KappaPos);
	tempL1.multiplyT(&tempL1);
	float r1 = sqrt(tempL1.max2());

	tempL1.set(&lLimit);
	tempL1.add(&Qtot);
	tempL1.subtract(&KappaPos2);
	tempL1.multiplyT(&tempL1);
	float r2 = sqrt(tempL1.max2());

	//std::cout << s1 << " " << s2 << " " << r1 << " " << r2 << std::endl;
	//std::cout << s1 << " " << s2 << " " << r1 << " " << r2 << std::endl;

	resXf = _ratioEps * (MAX(MAX(resXf, r1), r2));
	resX.set(0, iter, r1);
	resX.set(1, iter, r2);
	resX.set(2, iter, s1);
	resX.set(3, iter, s1);*/


	res->set(0, iter, resR);
	res->set(1, iter, resS);
	res->set(2, iter, resXf);

	/*if (iter != 0) {
		if (resR > _mu * resS) {
			_mu = _mu * 2;
			_rhog = _tau * _rhog;
			_at1 = _rhog;
			//std::cout << "rho augmente :" << _rhog << std::endl;
			//std::cout << iter << " " << resR << " " << resS << " " << resXf << std::endl;
		}
		else if (resS > _mu * resR) {// rho = rho / tau_inc;
			_mu = _mu * 2;
			_rhog = _rhog / _tau;
			_at1 = _rhog;
			//std::cout << "rho diminue :" << _rhog << std::endl;
			//std::cout << iter << " " << resR << " " << resS << " " << resXf << std::endl;
		}
		else if (resXf > _mu1 * resR) {
			_mu1 = _mu1*2;
			_rho1 = _tau * _rho1;
			Ap2b.multiply(_tau);
			Ap2.add(&Ap2a, &Ap2b);
			Ap12.add(&Ap1, &Ap2);
			//std::cout << "_rho1 augmente :" << _rho1 << std::endl;
			//std::cout << iter << " " << resR << " " << resS << " " << resXf << std::endl;
		}
		else if (resS > _mu1 * resXf) {
			_mu1 = _mu1 * 2;
			_rho1 = _rho1 / _tau;
			Ap2b.divide(_tau);
			Ap2.add(&Ap2a, &Ap2b);
			Ap12.add(&Ap1, &Ap2);
			//std::cout << "_rho1 diminue :" << _rho1 << std::endl;
			//std::cout << iter << " " << resR << " " << resS << " " << resXf << std::endl;
		}

		if ((r1 + r2) > 2 * _mu * (s1 + s2)) {

			_rho1 = _tau * _rho1;
			Ap2b.multiply(_tau);
			Ap2.add(&Ap2a, &Ap2b);
			Ap12.add(&Ap1, &Ap2);
			std::cout << "_rho1 augmente :" << _rho1 << std::endl;
			std::cout << iter << " " << resR << " " << resS << " " << r1 << " " << r2 << " " << s1 << " " << s2 << std::endl;
		}
		else if ((s1 + s2) > 2 * _mu * (r1 + r2)) {// rho = rho / tau_inc;
			_rho1 = _rho1 / _tau;
			Ap2b.divide(_tau);
			Ap2.add(&Ap2a, &Ap2b);
			Ap12.add(&Ap1, &Ap2);
			std::cout << "_rho1 diminue :" << _rho1 << std::endl;
			std::cout << iter << " " << resR << " " << resS << " " << r1 << " " << r2 << " " << s1 << " " << s2 << std::endl;
		}
	}*/
	

	return MAX(MAX(resXf, resS), resR);
}





void ADMMGPUConst4::display() {

	std::cout << _name << std::endl;
}


