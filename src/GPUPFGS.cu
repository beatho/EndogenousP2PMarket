#include "hip/hip_runtime.h"
#include "../head/GPUPFGS.CUh"


GPUPFGS::GPUPFGS(){}
GPUPFGS::~GPUPFGS(){}

void GPUPFGS::init(const StudyCase& cas, MatrixGPU* PQ, MatrixGPUD* PQD, bool useDouble)
{

#ifdef INSTRUMENTATION
    t1 = std::chrono::high_resolution_clock::now();
    timePerBlock = MatrixCPU(1, 9); // Fb0 : init, Fb1ab : Flu, Fb2abc: Tension , FB3 : puissance, Fb4 erreur, Fb0 mise � jour

    occurencePerBlock = MatrixCPU(1, 9);; //nb de fois utilis� pendant la simu
#endif // INSTRUMENTATION
    Nagent = cas.getNagent();
    Nbus = cas.getNBus();
    B2 = 2 * Nbus;
    N2 = 2 * Nagent;
    Nline = cas.getNLine(true); // ne doit pas �tre r�duit ici !!!
    BL2 = Nbus + 2 * Nline;
    Nconstraint = B2 + Nline;
    iterM = 5000;
    iter = 0;
    V0 = cas.getV0();
    theta0 = cas.gettheta0();
    v0 = V0 * cos(theta0);
    w0 = V0 * sin(theta0);
    _name = "Gauss-Seidel";
    _useDouble = useDouble;
    status = 0;
    
    //I = MatrixGPU(cas.getCoresBusAgentLin(), 1);
    CoresAgentBus = MatrixGPU(cas.getCoresAgentBusLin(), 1);
    CoresAgentBusBegin = MatrixGPU(cas.getCoresAgentBusLinBegin(), 1);
    NagentByBus = MatrixGPU(cas.getNagentByBus(), 1);
    //I.display(true);
    numBlock = Nbus;
    _useDouble = useDouble;


    

    CoresLineBus = MatrixGPU(cas.getCoresLineBus(true));
    _CoresVoiLin = MatrixGPU(cas.getCoresVoiLin());
    _CoresBusLin = MatrixGPU(cas.getCoresBusLin());
    _nLines = MatrixGPU(cas.getNLines());
    CoresTrans = MatrixGPU(BL2, 1, 0);

    int* decompte = new int[Nbus];
    for (int i = 0; i < Nbus; i++) {
        decompte[i] = 0;
    }

    
    for (int i = 0; i < Nbus; i++) {
        int begin = _CoresBusLin.get(i,0);
        for (int l = begin + 1; l < (begin + _nLines.get(i, 0)); l++) { // l = (j, i)
            int j = _CoresVoiLin.get(l, 0);
            CoresTrans.set(l, 0, _CoresBusLin.get(j, 0) + 1 + decompte[j]);
            decompte[j]++;
        }
    }
    CoresLineBusGPU = MatrixGPU(2, Nline);

    for (int lold = 0; lold < Nline; lold++) {
        int busTo = CoresLineBus.get(lold, 1);
        int busFrom = CoresLineBus.get(lold, 0);
        CoresLineBusGPU.set(0, lold, busFrom);
        CoresLineBusGPU.set(1, lold, busTo);
    }
    CoresLineBusGPU.transferGPU();
  
    DELETEA(decompte);

    _CoresVoiLin.transferGPU();
    _CoresBusLin.transferGPU();
    _nLines.transferGPU();
    CoresTrans.transferGPU();

    if (useDouble) {
        //BgridD = cas.getLineSuceptanceD();
        //GgridD = cas.getLineReactanceD();
        _BlinD = MatrixGPUD(cas.getBlinD(), 1);
        _GlinD = MatrixGPUD(cas.getGlinD(), 1);

        WD = MatrixGPUD(B2, 1, 0, 1);
        _PintermediateD = MatrixGPUD(BL2, 1, 0, 1);
        _QintermediateD = MatrixGPUD(BL2, 1, 0, 1);

        dWD = MatrixGPUD(B2, 1, 0, 1);
        dWD.preallocateReduction();
        ED = MatrixGPUD(B2, 1, 0, 1);

        initED << <numBlock, _blockSize >> > (ED._matrixGPU, theta0, V0, Nbus);
        // ED.display(true);

        dED = MatrixGPUD(B2, 1, 0, 1);
       
       
        W0D = MatrixGPUD(B2, 1, 0, 1);
        calculW0DBis(PQD);
        //W0D.display(true);

        /*Ggrid2Bgrid2 = MatrixCPU(Nbus, Nbus);
        for (int i = 0; i < Nbus; i++) {
            for (int j = 0; j < Nbus; j++) {
                Ggrid2Bgrid2.set(i, j, sqrt(GgridD.get(i, j) * GgridD.get(i, j) + BgridD.get(i, j) * BgridD.get(i, j)));
            }
        }*/
        
        RgridD = MatrixGPUD(Nbus, 1, 0, 1);
        XgridD = MatrixGPUD(Nbus, 1, 0, 1);
        RMGgridD = MatrixGPUD(BL2, 1, 0, 1);
        RPGgridD = MatrixGPUD(BL2, 1, 0, 1);
        //VectorResultD = MatrixGPUD(B2, 1, 0, 1);
        VoltageRealImD = MatrixGPUD(B2, 1, 0, 1);

        initEDCar <<<numBlock, _blockSize >> > (VoltageRealImD._matrixGPU, theta0, V0, Nbus);
        //VoltageRealImD.display(true);
        

        initRXD << <numBlock, _blockSize >> > (RgridD._matrixGPU, XgridD._matrixGPU, RMGgridD._matrixGPU, RPGgridD._matrixGPU, _GlinD._matrixGPU, _BlinD._matrixGPU, _CoresBusLin._matrixGPU, _nLines._matrixGPU);
        //initRXD2 << <numBlock, _blockSize >> > (RgridD._matrixGPU, XgridD._matrixGPU, RMGgridD._matrixGPU, RPGgridD._matrixGPU, _GlinD._matrixGPU, _BlinD._matrixGPU, _CoresBusLin._matrixGPU, _nLines._matrixGPU, CoresTrans._matrixGPU);


        calcW();
        //WD.display(true);
        dWD.subtract(&W0D, &WD);

        //VoltageRealImD.display(true);

        
       /*_GlinD.display(true);
       _BlinD.display(true);

       RgridD.display(true);
       XgridD.display(true);
       _CoresBusLin.display(true);
       _CoresVoiLin.display(true);
       RMGgridD.display(true);
       RPGgridD.display(true);*/

    }
    else {
        _Blin = MatrixGPU(cas.getBlin(), 1);
        _Glin = MatrixGPU(cas.getGlin(), 1);

        W = MatrixGPU(B2, 1, 0, 1);
        _Pintermediate = MatrixGPU(BL2, 1, 0, 1);
        _Qintermediate = MatrixGPU(BL2, 1, 0, 1);

        dW = MatrixGPU(B2, 1, 0, 1);
        dW.preallocateReduction();
        E = MatrixGPU(B2, 1, 0, 1);

        initE << <numBlock, _blockSize >> > (E._matrixGPU, theta0, V0, Nbus);
        //E.display(true);

        dE = MatrixGPUD(B2, 1, 0, 1);


        W0 = MatrixGPUD(B2, 1, 0, 1);
        calculW0Bis(PQ);
        //W0.display(true);

        /*Ggrid2Bgrid2 = MatrixCPU(Nbus, Nbus);
        for (int i = 0; i < Nbus; i++) {
            for (int j = 0; j < Nbus; j++) {
                Ggrid2Bgrid2.set(i, j, sqrt(GgridD.get(i, j) * GgridD.get(i, j) + BgridD.get(i, j) * BgridD.get(i, j)));
            }
        }*/

        Rgrid = MatrixGPU(Nbus, 1, 0, 1);
        Xgrid = MatrixGPU(Nbus, 1, 0, 1);
        RMGgrid = MatrixGPU(BL2, 1, 0, 1);
        RPGgrid = MatrixGPU(BL2, 1, 0, 1);
       // VectorResult = MatrixGPU(B2, 1, 0, 1);
        VoltageRealIm = MatrixGPU(B2, 1, 0, 1);

        initECar << <numBlock, _blockSize >> > (VoltageRealIm._matrixGPU, theta0, V0, Nbus);
        initRX << <numBlock, _blockSize >> > (Rgrid._matrixGPU, Xgrid._matrixGPU, RMGgrid._matrixGPU, RPGgrid._matrixGPU, _Glin._matrixGPU, _Blin._matrixGPU, _CoresBusLin._matrixGPU, _nLines._matrixGPU);

        
        /*_Glin.display(true);
        _Blin.display(true);

        Rgrid.display(true);
        Xgrid.display(true);
        _CoresBusLin.display(true);
        _CoresVoiLin.display(true);
        RMGgrid.display(true);
        RPGgrid.display(true);*/

        calcW();
        //W.display(true);
        dW.subtract(&W0, &W);


        //VoltageRealIm.display(true);

    }


    _Blin2 = MatrixGPU(cas.getBlin2(), 1);
    _Glin2 = MatrixGPU(cas.getGlin2(), 1);
    /*std::cout << " Bgrid : " << std::endl;
    Bgrid.display();
    std::cout << " Ggrid : " << std::endl;
    Ggrid.display(); */

    //std::cout << " E : " << std::endl;
    //E.display();
    
	// W0[2 * N] : puissance active et r�active au noeud (I*[P Q])
	// W[2 * N] : puissance obtenue par calcul � partir de E
	// dW[2 * N] : derive de puissance
	// E[2 * N] : angle puis tension [O et 1] pour l'init ?
	// dE[2 * N] : derive de angle puis tension
	// Jac[2 * N][2 * N] : jacobienne
	// Jac_inv[2 * N][2 * N]: inverse de la jacobienne
	
	// B[N][N], G[N][N] : caract�ristique des lignes entre les noeuds i et j

    /*G = MatrixCPU(Nconstraint, N2);
    Phi = MatrixCPU(Nline, 1);
    Y = MatrixCPU(Nconstraint, 1);
    tempLN2 = MatrixCPU(Nline, N2);
    JacPhiE = MatrixCPU(Nline, B2);
    tempB2N2 = MatrixCPU(B2, N2);*/
    



#ifdef INSTRUMENTATION
    hipDeviceSynchronize();
    t2 = std::chrono::high_resolution_clock::now();
    timePerBlock.increment(0, 0, (float) std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
    occurencePerBlock.increment(0, 0, 1);
#endif // INSTRUMENTATION
    //std::cout << numBlock << " " << _blockSize << std::endl;
    //std::cout << " fin init" << std::endl;

}



int GPUPFGS::calcVoltage()
{
    
    //calcE();
    //E.display();

  
   
    if (_useDouble) {  
        //VoltageRealImD.display(true);
#ifdef INSTRUMENTATION
        t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION
      
        switch (_blockSize) {
        case 512:
            calculVolDtStep1<512> << <numBlock, _blockSize , B2 * sizeof(double) >> > (VoltageRealImD._matrixGPU, W0D._matrixGPU, RgridD._matrixGPU, XgridD._matrixGPU, RMGgridD._matrixGPU, RPGgridD._matrixGPU, _CoresVoiLin._matrixGPU, _CoresBusLin._matrixGPU, _nLines._matrixGPU, Nbus);
            break;
        case 256:
            calculVolDtStep1<256> << <numBlock, _blockSize, B2 * sizeof(double) >> > (VoltageRealImD._matrixGPU, W0D._matrixGPU, RgridD._matrixGPU, XgridD._matrixGPU, RMGgridD._matrixGPU, RPGgridD._matrixGPU, _CoresVoiLin._matrixGPU, _CoresBusLin._matrixGPU, _nLines._matrixGPU, Nbus);
            break;
        case 128:
            calculVolDtStep1<128> << <numBlock, _blockSize, B2 * sizeof(double) >> > (VoltageRealImD._matrixGPU, W0D._matrixGPU, RgridD._matrixGPU, XgridD._matrixGPU, RMGgridD._matrixGPU, RPGgridD._matrixGPU, _CoresVoiLin._matrixGPU, _CoresBusLin._matrixGPU, _nLines._matrixGPU, Nbus);
            break;
        case 64:
            calculVolDtStep1< 64> << <numBlock, _blockSize, B2 * sizeof(double) >> > (VoltageRealImD._matrixGPU, W0D._matrixGPU, RgridD._matrixGPU, XgridD._matrixGPU, RMGgridD._matrixGPU, RPGgridD._matrixGPU, _CoresVoiLin._matrixGPU, _CoresBusLin._matrixGPU, _nLines._matrixGPU, Nbus);
            break;
        case 32:
            calculVolDtStep1< 32> << <numBlock, _blockSize, B2 * sizeof(double) >> > (VoltageRealImD._matrixGPU, W0D._matrixGPU, RgridD._matrixGPU, XgridD._matrixGPU, RMGgridD._matrixGPU, RPGgridD._matrixGPU, _CoresVoiLin._matrixGPU, _CoresBusLin._matrixGPU, _nLines._matrixGPU, Nbus);
            break;
        case 16:
            calculVolDtStep1< 16> << <numBlock, _blockSize, B2 * sizeof(double) >> > (VoltageRealImD._matrixGPU, W0D._matrixGPU, RgridD._matrixGPU, XgridD._matrixGPU, RMGgridD._matrixGPU, RPGgridD._matrixGPU, _CoresVoiLin._matrixGPU, _CoresBusLin._matrixGPU, _nLines._matrixGPU, Nbus);
            break;
        case  8:
            calculVolDtStep1<  8> << <numBlock, _blockSize, B2 * sizeof(double) >> > (VoltageRealImD._matrixGPU, W0D._matrixGPU, RgridD._matrixGPU, XgridD._matrixGPU, RMGgridD._matrixGPU, RPGgridD._matrixGPU, _CoresVoiLin._matrixGPU, _CoresBusLin._matrixGPU, _nLines._matrixGPU, Nbus);
            break;
        case  4:
            calculVolDtStep1<  4> << <numBlock, _blockSize, B2 * sizeof(double) >> > (VoltageRealImD._matrixGPU, W0D._matrixGPU, RgridD._matrixGPU, XgridD._matrixGPU, RMGgridD._matrixGPU, RPGgridD._matrixGPU, _CoresVoiLin._matrixGPU, _CoresBusLin._matrixGPU, _nLines._matrixGPU, Nbus);
            break;
        case  2:
            calculVolDtStep1<  2> << <numBlock, _blockSize, B2 * sizeof(double) >> > (VoltageRealImD._matrixGPU, W0D._matrixGPU, RgridD._matrixGPU, XgridD._matrixGPU, RMGgridD._matrixGPU, RPGgridD._matrixGPU, _CoresVoiLin._matrixGPU, _CoresBusLin._matrixGPU, _nLines._matrixGPU, Nbus);
            break;
        case  1:
            calculVolDtStep1<  1> << <numBlock, _blockSize, B2 * sizeof(double) >> > (VoltageRealImD._matrixGPU, W0D._matrixGPU, RgridD._matrixGPU, XgridD._matrixGPU, RMGgridD._matrixGPU, RPGgridD._matrixGPU, _CoresVoiLin._matrixGPU, _CoresBusLin._matrixGPU, _nLines._matrixGPU, Nbus);
            break;
        }
        //hipError_t c = hipPeekAtLastError();
        //std::cout << hipGetErrorString(c) << std::endl;
#ifdef INSTRUMENTATION
        hipDeviceSynchronize();
        t2 = std::chrono::high_resolution_clock::now();
        timePerBlock.increment(0, 3, (float) std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
        occurencePerBlock.increment(0, 3, 1);
        t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION
        //VoltageRealImD.display(true);
    
        //VoltageRealImD.display(true);
        
        //calculVoltDStep2 <<< 1, _blockSize, B2 * sizeof(double) >> > (VoltageRealImD._matrixGPU, RMGgridD._matrixGPU, RPGgridD._matrixGPU, _CoresVoiLin._matrixGPU, _CoresBusLin._matrixGPU, _nLines._matrixGPU, CoresTrans._matrixGPU, Nbus);

        calculVoltDStep2bis <<<1, _blockSize, 2 * (BL2 + Nbus) * sizeof(double) >>> (VoltageRealImD._matrixGPU, RMGgridD._matrixGPU, RPGgridD._matrixGPU, _CoresVoiLin._matrixGPU, _CoresBusLin._matrixGPU, _nLines._matrixGPU, CoresTrans._matrixGPU, Nbus, BL2);
        
        //VoltageRealImD.display(true);
        //std::cout << "----" << std::endl;
#ifdef INSTRUMENTATION
        hipDeviceSynchronize();
        t2 = std::chrono::high_resolution_clock::now();
        timePerBlock.increment(0, 4, (float) std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
        occurencePerBlock.increment(0, 4, 1);
#endif // INSTRUMENTATION

    }
    else {
        //VoltageRealIm.display(true);
#ifdef INSTRUMENTATION
        t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION
        switch (_blockSize) {
        case 512:
            calculVoltStep1<512> << <numBlock, _blockSize, B2 * sizeof(float) >> > (VoltageRealIm._matrixGPU, W0._matrixGPU, Rgrid._matrixGPU, Xgrid._matrixGPU, RMGgrid._matrixGPU, RPGgrid._matrixGPU, _CoresVoiLin._matrixGPU, _CoresBusLin._matrixGPU, _nLines._matrixGPU, Nbus);
            break;
        case 256:
            calculVoltStep1<256> << <numBlock, _blockSize, B2 * sizeof(float) >> > (VoltageRealIm._matrixGPU, W0._matrixGPU, Rgrid._matrixGPU, Xgrid._matrixGPU, RMGgrid._matrixGPU, RPGgrid._matrixGPU, _CoresVoiLin._matrixGPU, _CoresBusLin._matrixGPU, _nLines._matrixGPU, Nbus);
            break;
        case 128:
            calculVoltStep1<128> << <numBlock, _blockSize, B2 * sizeof(float) >> > (VoltageRealIm._matrixGPU, W0._matrixGPU, Rgrid._matrixGPU, Xgrid._matrixGPU, RMGgrid._matrixGPU, RPGgrid._matrixGPU, _CoresVoiLin._matrixGPU, _CoresBusLin._matrixGPU, _nLines._matrixGPU, Nbus);
            break;
        case 64:
            calculVoltStep1< 64> << <numBlock, _blockSize, B2 * sizeof(float) >> > (VoltageRealIm._matrixGPU, W0._matrixGPU, Rgrid._matrixGPU, Xgrid._matrixGPU, RMGgrid._matrixGPU, RPGgrid._matrixGPU, _CoresVoiLin._matrixGPU, _CoresBusLin._matrixGPU, _nLines._matrixGPU, Nbus);
            break;
        case 32:
            calculVoltStep1< 32> << <numBlock, _blockSize, B2 * sizeof(float) >> > (VoltageRealIm._matrixGPU, W0._matrixGPU, Rgrid._matrixGPU, Xgrid._matrixGPU, RMGgrid._matrixGPU, RPGgrid._matrixGPU, _CoresVoiLin._matrixGPU, _CoresBusLin._matrixGPU, _nLines._matrixGPU, Nbus);
            break;
        case 16:
            calculVoltStep1< 16> << <numBlock, _blockSize, B2 * sizeof(float) >> > (VoltageRealIm._matrixGPU, W0._matrixGPU, Rgrid._matrixGPU, Xgrid._matrixGPU, RMGgrid._matrixGPU, RPGgrid._matrixGPU, _CoresVoiLin._matrixGPU, _CoresBusLin._matrixGPU, _nLines._matrixGPU, Nbus);
            break;
        case  8:
            calculVoltStep1<  8> << <numBlock, _blockSize, B2 * sizeof(float) >> > (VoltageRealIm._matrixGPU, W0._matrixGPU, Rgrid._matrixGPU, Xgrid._matrixGPU, RMGgrid._matrixGPU, RPGgrid._matrixGPU, _CoresVoiLin._matrixGPU, _CoresBusLin._matrixGPU, _nLines._matrixGPU, Nbus);
            break;
        case  4:
            calculVoltStep1<  4> << <numBlock, _blockSize, B2 * sizeof(float) >> > (VoltageRealIm._matrixGPU, W0._matrixGPU, Rgrid._matrixGPU, Xgrid._matrixGPU, RMGgrid._matrixGPU, RPGgrid._matrixGPU, _CoresVoiLin._matrixGPU, _CoresBusLin._matrixGPU, _nLines._matrixGPU, Nbus);
            break;
        case  2:
            calculVoltStep1<  2> << <numBlock, _blockSize, B2 * sizeof(float) >> > (VoltageRealIm._matrixGPU, W0._matrixGPU, Rgrid._matrixGPU, Xgrid._matrixGPU, RMGgrid._matrixGPU, RPGgrid._matrixGPU, _CoresVoiLin._matrixGPU, _CoresBusLin._matrixGPU, _nLines._matrixGPU, Nbus);
            break;
        case  1:
            calculVoltStep1<  1> << <numBlock, _blockSize, B2 * sizeof(float) >> > (VoltageRealIm._matrixGPU, W0._matrixGPU, Rgrid._matrixGPU, Xgrid._matrixGPU, RMGgrid._matrixGPU, RPGgrid._matrixGPU, _CoresVoiLin._matrixGPU, _CoresBusLin._matrixGPU, _nLines._matrixGPU, Nbus);
            break;
    }
#ifdef INSTRUMENTATION
        hipDeviceSynchronize();
        t2 = std::chrono::high_resolution_clock::now();
        timePerBlock.increment(0, 3, (float) std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
        occurencePerBlock.increment(0, 3, 1);
        t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION

       // VoltageRealIm.display(true);

        
        //calculVoltStep2 << <1, _blockSize, B2 * sizeof(float)>>> (VoltageRealIm._matrixGPU, RMGgrid._matrixGPU, RPGgrid._matrixGPU, _CoresVoiLin._matrixGPU, _CoresBusLin._matrixGPU, _nLines._matrixGPU, CoresTrans._matrixGPU, Nbus);

        calculVoltStep2bis <<<1, _blockSize, 2 * (BL2 + Nbus) * sizeof(float) >> > (VoltageRealIm._matrixGPU, RMGgrid._matrixGPU, RPGgrid._matrixGPU, _CoresVoiLin._matrixGPU, _CoresBusLin._matrixGPU, _nLines._matrixGPU, CoresTrans._matrixGPU, Nbus, BL2);
        
        //VoltageRealIm.display(true);
        //std::cout << "----" << std::endl;
#ifdef INSTRUMENTATION
        hipDeviceSynchronize();
        t2 = std::chrono::high_resolution_clock::now();
        timePerBlock.increment(0, 4, (float) std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
        occurencePerBlock.increment(0, 4, 1);
#endif // INSTRUMENTATION

    }
    return 0;

}


void GPUPFGS::calcE()
{
    if (_useDouble) {
        calcEGPUD << <numBlock, _blockSize >> > (ED._matrixGPU, VoltageRealImD._matrixGPU, Nbus);
    }
    else {
        calcEGPU << <numBlock, _blockSize >> > (E._matrixGPU, VoltageRealIm._matrixGPU, Nbus);
    }
   

}

void GPUPFGS::calcW(bool end)
{
    if (_useDouble) {

        calcWinterCarD << <numBlock, _blockSize, B2 * sizeof(double) >> > (_PintermediateD._matrixGPU, _QintermediateD._matrixGPU, VoltageRealImD._matrixGPU, _GlinD._matrixGPU, _BlinD._matrixGPU, _CoresVoiLin._matrixGPU, _CoresBusLin._matrixGPU, _nLines._matrixGPU, Nbus);

        if (!end) { // pendant simu, la puissance � ce noeud est libre
            switch (_blockSize) {
            case 512:
                calcWGPUD<512> << <numBlock, _blockSize >> > (WD._matrixGPU, W0D._matrixGPU, _PintermediateD._matrixGPU, _QintermediateD._matrixGPU, _CoresBusLin._matrixGPU, _nLines._matrixGPU, Nbus);
                break;
            case 256:
                calcWGPUD<256> << <numBlock, _blockSize >> > (WD._matrixGPU, W0D._matrixGPU, _PintermediateD._matrixGPU, _QintermediateD._matrixGPU, _CoresBusLin._matrixGPU, _nLines._matrixGPU, Nbus);
                break;
            case 128:
                calcWGPUD<128> << <numBlock, _blockSize >> > (WD._matrixGPU, W0D._matrixGPU, _PintermediateD._matrixGPU, _QintermediateD._matrixGPU, _CoresBusLin._matrixGPU, _nLines._matrixGPU, Nbus);
                break;
            case 64:
                calcWGPUD< 64> << <numBlock, _blockSize >> > (WD._matrixGPU, W0D._matrixGPU, _PintermediateD._matrixGPU, _QintermediateD._matrixGPU, _CoresBusLin._matrixGPU, _nLines._matrixGPU, Nbus);
                break;
            case 32:
                calcWGPUD< 32> << <numBlock, _blockSize >> > (WD._matrixGPU, W0D._matrixGPU, _PintermediateD._matrixGPU, _QintermediateD._matrixGPU, _CoresBusLin._matrixGPU, _nLines._matrixGPU, Nbus);
                break;
            case 16:
                calcWGPUD< 16> << <numBlock, _blockSize >> > (WD._matrixGPU, W0D._matrixGPU, _PintermediateD._matrixGPU, _QintermediateD._matrixGPU, _CoresBusLin._matrixGPU, _nLines._matrixGPU, Nbus);
                break;
            case  8:
                calcWGPUD<  8> << <numBlock, _blockSize >> > (WD._matrixGPU, W0D._matrixGPU, _PintermediateD._matrixGPU, _QintermediateD._matrixGPU, _CoresBusLin._matrixGPU, _nLines._matrixGPU, Nbus);
                break;
            case  4:
                calcWGPUD<  4> << <numBlock, _blockSize >> > (WD._matrixGPU, W0D._matrixGPU, _PintermediateD._matrixGPU, _QintermediateD._matrixGPU, _CoresBusLin._matrixGPU, _nLines._matrixGPU, Nbus);
                break;
            case  2:
                calcWGPUD<  2> << <numBlock, _blockSize >> > (WD._matrixGPU, W0D._matrixGPU, _PintermediateD._matrixGPU, _QintermediateD._matrixGPU, _CoresBusLin._matrixGPU, _nLines._matrixGPU, Nbus);
                break;
            case  1:
                calcWGPUD<  1> << <numBlock, _blockSize >> > (WD._matrixGPU, W0D._matrixGPU, _PintermediateD._matrixGPU, _QintermediateD._matrixGPU, _CoresBusLin._matrixGPU, _nLines._matrixGPU, Nbus);
                break;
            }
        }
        else {
            switch (_blockSize) {
            case 512:
                calcWGPUD<512> << <numBlock, _blockSize >> > (WD._matrixGPU, _PintermediateD._matrixGPU, _QintermediateD._matrixGPU, _CoresBusLin._matrixGPU, _nLines._matrixGPU, Nbus);
                break;
            case 256:
                calcWGPUD<256> << <numBlock, _blockSize >> > (WD._matrixGPU, _PintermediateD._matrixGPU, _QintermediateD._matrixGPU, _CoresBusLin._matrixGPU, _nLines._matrixGPU, Nbus);
                break;
            case 128:
                calcWGPUD<128> << <numBlock, _blockSize >> > (WD._matrixGPU, _PintermediateD._matrixGPU, _QintermediateD._matrixGPU, _CoresBusLin._matrixGPU, _nLines._matrixGPU, Nbus);
                break;
            case 64:
                calcWGPUD< 64> << <numBlock, _blockSize >> > (WD._matrixGPU, _PintermediateD._matrixGPU, _QintermediateD._matrixGPU, _CoresBusLin._matrixGPU, _nLines._matrixGPU, Nbus);
                break;
            case 32:
                calcWGPUD< 32> << <numBlock, _blockSize >> > (WD._matrixGPU, _PintermediateD._matrixGPU, _QintermediateD._matrixGPU, _CoresBusLin._matrixGPU, _nLines._matrixGPU, Nbus);
                break;
            case 16:
                calcWGPUD< 16> << <numBlock, _blockSize >> > (WD._matrixGPU, _PintermediateD._matrixGPU, _QintermediateD._matrixGPU, _CoresBusLin._matrixGPU, _nLines._matrixGPU, Nbus);
                break;
            case  8:
                calcWGPUD<  8> << <numBlock, _blockSize >> > (WD._matrixGPU, _PintermediateD._matrixGPU, _QintermediateD._matrixGPU, _CoresBusLin._matrixGPU, _nLines._matrixGPU, Nbus);
                break;
            case  4:
                calcWGPUD<  4> << <numBlock, _blockSize >> > (WD._matrixGPU, _PintermediateD._matrixGPU, _QintermediateD._matrixGPU, _CoresBusLin._matrixGPU, _nLines._matrixGPU, Nbus);
                break;
            case  2:
                calcWGPUD<  2> << <numBlock, _blockSize >> > (WD._matrixGPU, _PintermediateD._matrixGPU, _QintermediateD._matrixGPU, _CoresBusLin._matrixGPU, _nLines._matrixGPU, Nbus);
                break;
            case  1:
                calcWGPUD<  1> << <numBlock, _blockSize >> > (WD._matrixGPU, _PintermediateD._matrixGPU, _QintermediateD._matrixGPU, _CoresBusLin._matrixGPU, _nLines._matrixGPU, Nbus);
                break;
            }

        }
    }
    else {
        /* E.display(true);
         _Glin.display(true);
         _Blin.display(true);
         std::cout << "------------" << std::endl;
         _CoresVoiLin.display(true);
         _CoresBusLin.display(true);
         _nLines.display(true);
         std::cout << "------------" << std::endl;*/

        calcWinterCar << <numBlock, _blockSize, B2 * sizeof(float) >> > (_Pintermediate._matrixGPU, _Qintermediate._matrixGPU, VoltageRealIm._matrixGPU, _Glin._matrixGPU, _Blin._matrixGPU, _CoresVoiLin._matrixGPU, _CoresBusLin._matrixGPU, _nLines._matrixGPU, Nbus);

        /*_Pintermediate.display(true);
        _Qintermediate.display(true);*/
        if (!end) { // pendant simu, la puissance � ce noeud est libre
            switch (_blockSize) {
            case 512:
                calcWGPU<512> << <numBlock, _blockSize >> > (W._matrixGPU, W0._matrixGPU, _Pintermediate._matrixGPU, _Qintermediate._matrixGPU, _CoresBusLin._matrixGPU, _nLines._matrixGPU, Nbus);
                break;
            case 256:
                calcWGPU<256> << <numBlock, _blockSize >> > (W._matrixGPU, W0._matrixGPU, _Pintermediate._matrixGPU, _Qintermediate._matrixGPU, _CoresBusLin._matrixGPU, _nLines._matrixGPU, Nbus);
                break;
            case 128:
                calcWGPU<128> << <numBlock, _blockSize >> > (W._matrixGPU, W0._matrixGPU, _Pintermediate._matrixGPU, _Qintermediate._matrixGPU, _CoresBusLin._matrixGPU, _nLines._matrixGPU, Nbus);
                break;
            case 64:
                calcWGPU< 64> << <numBlock, _blockSize >> > (W._matrixGPU, W0._matrixGPU, _Pintermediate._matrixGPU, _Qintermediate._matrixGPU, _CoresBusLin._matrixGPU, _nLines._matrixGPU, Nbus);
                break;
            case 32:
                calcWGPU< 32> << <numBlock, _blockSize >> > (W._matrixGPU, W0._matrixGPU, _Pintermediate._matrixGPU, _Qintermediate._matrixGPU, _CoresBusLin._matrixGPU, _nLines._matrixGPU, Nbus);
                break;
            case 16:
                calcWGPU< 16> << <numBlock, _blockSize >> > (W._matrixGPU, W0._matrixGPU, _Pintermediate._matrixGPU, _Qintermediate._matrixGPU, _CoresBusLin._matrixGPU, _nLines._matrixGPU, Nbus);
                break;
            case  8:
                calcWGPU<  8> << <numBlock, _blockSize >> > (W._matrixGPU, W0._matrixGPU, _Pintermediate._matrixGPU, _Qintermediate._matrixGPU, _CoresBusLin._matrixGPU, _nLines._matrixGPU, Nbus);
                break;
            case  4:
                calcWGPU<  4> << <numBlock, _blockSize >> > (W._matrixGPU, W0._matrixGPU, _Pintermediate._matrixGPU, _Qintermediate._matrixGPU, _CoresBusLin._matrixGPU, _nLines._matrixGPU, Nbus);
                break;
            case  2:
                calcWGPU<  2> << <numBlock, _blockSize >> > (W._matrixGPU, W0._matrixGPU, _Pintermediate._matrixGPU, _Qintermediate._matrixGPU, _CoresBusLin._matrixGPU, _nLines._matrixGPU, Nbus);
                break;
            case  1:
                calcWGPU<  1> << <numBlock, _blockSize >> > (W._matrixGPU, W0._matrixGPU, _Pintermediate._matrixGPU, _Qintermediate._matrixGPU, _CoresBusLin._matrixGPU, _nLines._matrixGPU, Nbus);
                break;
            }
        }
        else {
           // _Pintermediate.display(true);
            switch (_blockSize) {
            case 512:
                calcWGPU<512> << <numBlock, _blockSize >> > (W._matrixGPU, _Pintermediate._matrixGPU, _Qintermediate._matrixGPU, _CoresBusLin._matrixGPU, _nLines._matrixGPU, Nbus);
                break;
            case 256:
                calcWGPU<256> << <numBlock, _blockSize >> > (W._matrixGPU, _Pintermediate._matrixGPU, _Qintermediate._matrixGPU, _CoresBusLin._matrixGPU, _nLines._matrixGPU, Nbus);
                break;
            case 128:
                calcWGPU<128> << <numBlock, _blockSize >> > (W._matrixGPU, _Pintermediate._matrixGPU, _Qintermediate._matrixGPU, _CoresBusLin._matrixGPU, _nLines._matrixGPU, Nbus);
                break;
            case 64:
                calcWGPU< 64> << <numBlock, _blockSize >> > (W._matrixGPU, _Pintermediate._matrixGPU, _Qintermediate._matrixGPU, _CoresBusLin._matrixGPU, _nLines._matrixGPU, Nbus);
                break;
            case 32:
                calcWGPU< 32> << <numBlock, _blockSize >> > (W._matrixGPU, _Pintermediate._matrixGPU, _Qintermediate._matrixGPU, _CoresBusLin._matrixGPU, _nLines._matrixGPU, Nbus);
                break;
            case 16:
                calcWGPU< 16> << <numBlock, _blockSize >> > (W._matrixGPU, _Pintermediate._matrixGPU, _Qintermediate._matrixGPU, _CoresBusLin._matrixGPU, _nLines._matrixGPU, Nbus);
                break;
            case  8:
                calcWGPU<  8> << <numBlock, _blockSize >> > (W._matrixGPU, _Pintermediate._matrixGPU, _Qintermediate._matrixGPU, _CoresBusLin._matrixGPU, _nLines._matrixGPU, Nbus);
                break;
            case  4:
                calcWGPU<  4> << <numBlock, _blockSize >> > (W._matrixGPU, _Pintermediate._matrixGPU, _Qintermediate._matrixGPU, _CoresBusLin._matrixGPU, _nLines._matrixGPU, Nbus);
                break;
            case  2:
                calcWGPU<  2> << <numBlock, _blockSize >> > (W._matrixGPU, _Pintermediate._matrixGPU, _Qintermediate._matrixGPU, _CoresBusLin._matrixGPU, _nLines._matrixGPU, Nbus);
                break;
            case  1:
                calcWGPU<  1> << <numBlock, _blockSize >> > (W._matrixGPU, _Pintermediate._matrixGPU, _Qintermediate._matrixGPU, _CoresBusLin._matrixGPU, _nLines._matrixGPU, Nbus);
                break;
            }
            //W.display(true);
        }

    }



}

void GPUPFGS::setE(MatrixGPU* Enew)
{
#ifdef INSTRUMENTATION
    t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION
    E = *Enew;
    if (!E.getPos()) {
        E.transferGPU();
    }
    if (_useDouble) {
        E.toMatGPUD(ED);
        initEDCar << <numBlock, _blockSize >> > (VoltageRealImD._matrixGPU, ED._matrixGPU, Nbus);
    }
    else {
        initECar << <numBlock, _blockSize >> > (VoltageRealIm._matrixGPU, E._matrixGPU, Nbus);
    }
#ifdef INSTRUMENTATION
    t2 = std::chrono::high_resolution_clock::now();
    timePerBlock.increment(0, 8, (float) std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
    occurencePerBlock.increment(0, 8, 1);
    t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION
    calcW();
#ifdef INSTRUMENTATION
    t2 = std::chrono::high_resolution_clock::now();
    timePerBlock.increment(0, 6, (float) std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
    occurencePerBlock.increment(0, 6, 1);
    t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION
    if (_useDouble) {
        dWD.subtract(&W0D, &WD);
        err = dWD.max2(); //err = ||dW||
    }
    else {
        dW.subtract(&W0, &W);
        err = dW.max2(); //err = ||dW||
    }
#ifdef INSTRUMENTATION
    t2 = std::chrono::high_resolution_clock::now();
    timePerBlock.increment(0, 7, (float) std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
    occurencePerBlock.increment(0, 7, 1);
#endif // INSTRUMENTATION

}

void GPUPFGS::setE(MatrixGPUD* Enew)
{
#ifdef INSTRUMENTATION
    t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION
    ED = *Enew;
    if (!ED.getPos()) {
        ED.transferGPU();
    }
    if (_useDouble) {
        initEDCar << <numBlock, _blockSize >> > (VoltageRealImD._matrixGPU, ED._matrixGPU, Nbus);
    }
    else {
        E = ED;
        initECar << <numBlock, _blockSize >> > (VoltageRealIm._matrixGPU, E._matrixGPU, Nbus);
    }
    
#ifdef INSTRUMENTATION
    t2 = std::chrono::high_resolution_clock::now();
    timePerBlock.increment(0, 8, (float) std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
    occurencePerBlock.increment(0, 8, 1);
    t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION
    calcW();
#ifdef INSTRUMENTATION
    t2 = std::chrono::high_resolution_clock::now();
    timePerBlock.increment(0, 6, (float) std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
    occurencePerBlock.increment(0, 6, 1);
    t1 = std::chrono::high_resolution_clock::now();
#endif // INSTRUMENTATION
    if (_useDouble) {
        dWD.subtract(&W0D, &WD);
        err = dWD.max2(); //err = ||dW||
    }
    else {
        dW.subtract(&W0, &W);
        err = dW.max2(); //err = ||dW||
    }
#ifdef INSTRUMENTATION
    t2 = std::chrono::high_resolution_clock::now();
    timePerBlock.increment(0, 7, (float) std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count());
    occurencePerBlock.increment(0, 7, 1);
#endif // INSTRUMENTATION�
}



__global__ void initEDCar(double* VoltageRealImD, double* ED, int B) {


    int thIdx = threadIdx.x + blockDim.x * blockIdx.x;
    int size = gridDim.x * blockDim.x;
    for (int i = thIdx; i < B; i += size) {
        double V0 = ED[i + B];
        double theta0 = ED[i];

        VoltageRealImD[i] = V0 * cos(theta0);
        VoltageRealImD[i + B] = V0 * sin(theta0);
    }
}
__global__ void initEDCar(double* VoltageRealImD, double v0, double w0, int B) {


    int thIdx = threadIdx.x + blockDim.x * blockIdx.x;
    int size = gridDim.x * blockDim.x;
    for (int i = thIdx; i < B; i += size) {

        VoltageRealImD[i] = v0;
        VoltageRealImD[i + B] = w0;
    }
}



__global__ void initRX(float* Rgrid, float* Xgrid, float* RMGgrid, float* RPGgrid, float* Glin, float* Blin, float* CoresBusLin, float* nLines) {
    int index = threadIdx.x;
    int step = blockDim.x;
    int i = blockIdx.x;
    
    int begin = CoresBusLin[i];
    int end = begin + nLines[i];
   


    float norm = Glin[begin] * Glin[begin] + Blin[begin] * Blin[begin];
    float r =    Glin[begin] / norm;
    float x =   -Blin[begin] / norm;

    for (int l = begin + index + 1; l < end; l += step) {
       
        float m = Glin[l] * r - Blin[l] * x;
        float n = Blin[l] * r + Glin[l] * x;

        RMGgrid[l] = m;
        RPGgrid[l] = n;

    } 
    if (index == 0) {
        Rgrid[i] = r;
        Xgrid[i] = x;
    }
}
__global__ void initRXD(double* RgridD, double* XgridD, double* RMGgridD, double* RPGgridD, double* GlinD, double* BlinD, float* CoresBusLin, float* nLines) {
    int index = threadIdx.x;
    int step = blockDim.x;
    int i = blockIdx.x;

    int begin = CoresBusLin[i];
    int end = begin + nLines[i];


    double norm = (GlinD[begin] * GlinD[begin] + BlinD[begin] * BlinD[begin]);
    double r = GlinD[begin] / norm; // Re(1/Y)
    double x = -BlinD[begin] / norm; // Im(1/Y)

  
    
    for (int l = begin + index + 1; l < end; l += step) {

        float m = GlinD[l] * r - BlinD[l] * x;
        float n = BlinD[l] * r + GlinD[l] * x;

        RMGgridD[l] = m; //Re(Yij/Yii)
        RPGgridD[l] = n; //Im(Yij/Yii)

    }  
    if (index == 0) {
        RgridD[i] = r;
        XgridD[i] = x;
    }
}





/*
 for (int i = 1; i < Nbus; i++) {
            double vi = VoltageRealImD.get(i, 0);
            double wi = VoltageRealImD.get(i + Nbus, 0);
            double norm = vi * vi + wi * wi;
            double c = (W0D.get(i, 0) * vi + W0D.get(i + Nbus, 0) * wi) / norm;
            double d = (W0D.get(i, 0) * wi - W0D.get(i + Nbus, 0) * vi) / norm;
            double sum1 = c * RgridD.get(i, 0) - d * XgridD.get(i, 0);
            double sum2 = d * RgridD.get(i, 0) + c * XgridD.get(i, 0);
            
        }

*/
template <unsigned int blockSize>
__global__ void calculVoltStep1(float* VoltageRealIm, float* W0, float* Rgrid, float* Xgrid, float* RMGgrid, float* RPGgrid, float* CoresVoiLin, float* CoresBusLin, float* nLines, int B) {
    
    __shared__ float shArr[blockSize];
    __shared__ float shArr2[blockSize];
    extern __shared__ float shE[];
    int thIdx = threadIdx.x;
    int i = blockIdx.x;
    int step = blockSize;
    int begin = CoresBusLin[i];
    int end = begin + nLines[i];
    int B2 = 2 * B;

    if (i != 0) {
        for (int n = thIdx; n < B2; n += step)
        {
            shE[n] = VoltageRealIm[n];
        }
        __syncthreads();
        float sum = 0;
        float sum2 = 0;
        for (int l = begin + thIdx + 1; l < end; l += step) {
            int k = CoresVoiLin[l];
            if (k > i) {
                sum  -= (RMGgrid[l] * shE[k] - RPGgrid[l] * shE[k + B]);
                sum2 -= (RPGgrid[l] * shE[k] + RMGgrid[l] * shE[k + B]);
            }
        }


        shArr[thIdx] = sum;
        shArr2[thIdx] = sum2;
        __syncthreads();

        if (blockSize >= 512) { 
            if (thIdx < 256) {
                shArr[thIdx] += shArr[thIdx + 256]; 
                shArr2[thIdx] += shArr2[thIdx + 256];
            } 
        __syncthreads();
        }
        if (blockSize >= 256) {
            if (thIdx < 128) {
                shArr[thIdx] += shArr[thIdx + 128]; 
                shArr2[thIdx] += shArr2[thIdx + 128];
            } 
            __syncthreads();
        }
        if (blockSize >= 128) {
            if (thIdx < 64) {
                shArr[thIdx] += shArr[thIdx + 64]; 
                shArr[thIdx] += shArr2[thIdx + 64];
            } __syncthreads(); 
        }
        if (thIdx < 32) {
            warpReduce<blockSize>(shArr, thIdx);
            warpReduce<blockSize>(shArr2, thIdx);
        }
        if (thIdx == 0) {
            float vi = shE[i];
            float wi = shE[i + B];
            float r = Rgrid[i];
            float x = Xgrid[i];
            float W0_local = W0[i];
            float W0B_local = W0[i + B];

            float norm = vi * vi + wi * wi;
            float c = (W0_local * vi + W0B_local * wi) / norm;
            float d = (W0_local * wi - W0B_local * vi) / norm;


       
            VoltageRealIm[i] = shArr[0] + c * r - d * x;
            VoltageRealIm[i + B] = shArr2[0] + d * r + c * x;
        }
    }
}


template <unsigned int blockSize>
__global__ void calculVolDtStep1(double* VoltageRealImD, double* W0, double* Rgrid, double* Xgrid, double* RMGgrid, double* RPGgrid, float* CoresVoiLin, float* CoresBusLin, float* nLines, int B) {

    __shared__ double shArr[blockSize];
    __shared__ double shArr2[blockSize];
    extern __shared__ double shED[];
    int thIdx = threadIdx.x;
    int i = blockIdx.x;
    int step = blockSize;
    int begin = CoresBusLin[i];
    int end = begin + nLines[i];
    int B2 = 2 * B;

    if (i != 0) {
        for (int n = thIdx; n < B2; n += step)
        {
            shED[n] = VoltageRealImD[n];
        }
        __syncthreads();
        double sum = 0;
        double sum2 = 0;
        for (int l = begin + thIdx; l < end; l += step) {
            int k = CoresVoiLin[l];
            if (k > i) {
                sum += -(RMGgrid[l] * shED[k] - RPGgrid[l] * shED[k + B]);
                sum2 += -(RPGgrid[l] * shED[k] + RMGgrid[l] * shED[k + B]);
            }
           
        }


        shArr[thIdx] = sum;
        shArr2[thIdx] = sum2;
        __syncthreads();

        if (blockSize >= 512) {
            if (thIdx < 256) {
                shArr[thIdx] += shArr[thIdx + 256];
                shArr2[thIdx] += shArr2[thIdx + 256];
            }
            __syncthreads();
        }
        if (blockSize >= 256) {
            if (thIdx < 128) {
                shArr[thIdx] += shArr[thIdx + 128];
                shArr2[thIdx] += shArr2[thIdx + 128];
            }
            __syncthreads();
        }
        if (blockSize >= 128) {
            if (thIdx < 64) {
                shArr[thIdx] += shArr[thIdx + 64];
                shArr[thIdx] += shArr2[thIdx + 64];
            } __syncthreads();
        }
        if (thIdx < 32) {
            warpReduce<blockSize>(shArr, thIdx);
            warpReduce<blockSize>(shArr2, thIdx);
        }
        if (thIdx == 0) {
            double vi = shED[i];
            double wi = shED[i + B];
            double r = Rgrid[i];
            double x = Xgrid[i];
            double W0_local = W0[i];
            double W0B_local = W0[i + B];

            double norm = vi * vi + wi * wi;
            double c = (W0_local * vi + W0B_local * wi) / norm;
            double d = (W0_local * wi - W0B_local * vi) / norm;



            VoltageRealImD[i] = shArr[0] + c * r - d * x;
            VoltageRealImD[i + B] = shArr2[0] + d * r + c * x;
        }
    }
    
}


/*
for (int iter = 0; iter < Nbus-1; iter++) {
            for (int i = iter + 1; i < Nbus; i++) {
                double db1 = RMGgridD.get(i, iter) * VoltageRealImD.get(iter, 0) - RPGgridD.get(i, iter) * VoltageRealImD.get(iter + Nbus, 0);
                double db2 = RPGgridD.get(i, iter) * VoltageRealImD.get(iter, 0) + RMGgridD.get(i, iter) * VoltageRealImD.get(iter + Nbus, 0);

                VoltageRealImD.increment(i, 0, -db1);
                VoltageRealImD.increment(i + Nbus, 0, -db2);
            }
        }
*
*/


__global__ void calculVoltStep2(float* VoltageRealIm, float* RMGgrid, float* RPGgrid, float* CoresVoiLin, float* CoresBusLin, float* nLines, float* CoresTrans, int B) {

   /* int thIdx = threadIdx.x;
    int i = blockIdx.x;
    int size = blockDim.x;
    int begin = CoresBusLin[i]; // k = CoresBusLin[iter]; !!!
    int end = begin + nLines[i]; // k + nLines[iter]
    // il ne faut pas Ypj/Ypp mais bien Yjp/Yjj, donc il faut savoir quel voisin est p... 


   if(i>iter){
        for (int voisin = thIdx + begin + 1; voisin < end; voisin += size) {
            int p = CoresVoiLin[voisin];
           
            if (p == iter ) { // pour trouver quel indice est p, c'est plut�t nul
                
                float db1 = RMGgrid[voisin] * VoltageRealIm[iter] - RPGgrid[voisin] * VoltageRealIm[iter + B];
                float db2 = RPGgrid[voisin] * VoltageRealIm[iter] + RMGgrid[voisin] * VoltageRealIm[iter + B];

                VoltageRealIm[i] = VoltageRealIm[i] - db1;
                VoltageRealIm[i + B] = VoltageRealIm[i + B] - db2;
            }
        }
   }*/
    
    int thIdx = threadIdx.x;
    //int i = blockIdx.x; un seul bloc
    int size = blockDim.x;

    extern __shared__ float Voltage[];

    for (int k = thIdx; k < 2 * B; k += size) {
        Voltage[k] = VoltageRealIm[k];
    }
    __syncthreads();


    for (int iter = 0; iter < B - 1; iter++) {
        int begin = CoresBusLin[iter]; // k = CoresBusLin[iter]; !!!
        int end = begin + nLines[iter]; // k + nLines[iter]
        float ei = Voltage[iter];
        float fi = Voltage[iter + B];

        for (int l = thIdx + begin + 1; l < end; l += size) { // voisin
            int j = CoresVoiLin[l];
            
            if (j > iter) {
                int lTrans = CoresTrans[l]; // acc�s pas du tout coalescent !!!

                float ri = RMGgrid[lTrans]; // acc�s pas du tout coalescent mais c'est sur la m�moire partag�
                float li = RPGgrid[lTrans];

                float db1 = ri * ei - li * fi;
                float db2 = li * ei + ri * fi;

                Voltage[j] = Voltage[j] - db1;
                Voltage[j + B] = Voltage[j + B] - db2;
            }
        }
        __syncthreads();
    }

    for (int k = thIdx; k < 2 * B; k += size) {
        VoltageRealIm[k] = Voltage[k];
    }
    __syncthreads();


}


__global__ void calculVoltDStep2(double* VoltageRealIm, double* RMGgrid, double* RPGgrid, float* CoresVoiLin, float* CoresBusLin, float* nLines, float* CoresTrans, int B) {

    int thIdx = threadIdx.x;
    //int i = blockIdx.x; un seul bloc
    int size = blockDim.x;

    extern __shared__ double VoltageD[];

    for (int k = thIdx; k < 2 * B; k += size) {
        VoltageD[k] = VoltageRealIm[k];
    }
    __syncthreads();


    for (int iter = 0; iter < B - 1; iter++) {
        int begin = CoresBusLin[iter]; // k = CoresBusLin[iter]; !!!
        int end = begin + nLines[iter]; // k + nLines[iter]
        double ei = VoltageD[iter];
        double fi = VoltageD[iter + B];
        __syncthreads();
        for (int l = thIdx + begin + 1; l < end; l += size) { // voisin
            int j = CoresVoiLin[l];

            if (j > iter) {
                int lTrans = CoresTrans[l]; // acc�s coalescent !!!

                double ri = RMGgrid[lTrans]; // acc�s pas du tout coalescent mais c'est sur la m�moire partag�
                double li = RPGgrid[lTrans];

                double db1 = ri * ei - li * fi;
                double db2 = li * ei + ri * fi;

                VoltageD[j] = VoltageD[j] - db1;
                VoltageD[j + B] = VoltageD[j + B] - db2;
            }
        }
        __syncthreads();
    }
    for (int k = thIdx; k < 2 * B; k += size) {
        VoltageRealIm[k] = VoltageD[k];
    }
    __syncthreads();
    
}

__global__ void calculVoltStep2bis(float* VoltageRealIm, float* RMGgrid, float* RPGgrid, float* CoresVoiLin, float* CoresBusLin, float* nLines, float* CoresTrans, int B, int BL2) {

    
    int thIdx = threadIdx.x;
    //int i = blockIdx.x; un seul bloc
    int size = blockDim.x;
   

    extern __shared__ float RI[];
    
    float* Voltage = &RI[2 * BL2];
    for (int l = thIdx; l < BL2; l += size) { // coalecent
        RI[l] = RMGgrid[l];
        RI[l + BL2] = RPGgrid[l];
    }
    for (int k = thIdx; k < 2 * B; k += size) {
        Voltage[k] = VoltageRealIm[k];
    }
    __syncthreads();


    for (int iter = 0; iter < B - 1; iter++) {
        int begin = CoresBusLin[iter]; // k = CoresBusLin[iter]; !!!
        int end = begin + nLines[iter]; // k + nLines[iter]

        float ei = Voltage[iter];
        float fi = Voltage[iter + B];

        for (int l = thIdx + begin + 1; l < end; l += size) { // voisin
            int j = CoresVoiLin[l];

            if (j > iter) {
                int lTrans = CoresTrans[l];

                float ri = RI[lTrans]; // acc�s pas du tout coalescent mais c'est sur la m�moire partag�
                float li = RI[lTrans + BL2];


                float db1 = ri * ei - li * fi;
                float db2 = li * ei + ri * fi;

                Voltage[j] = Voltage[j] - db1;
                Voltage[j + B] = Voltage[j + B] - db2;
            }
        }
        __syncthreads();
    }
    for (int k = thIdx; k < 2 * B; k += size) {
        VoltageRealIm[k] = Voltage[k];
    }
    __syncthreads();

}


__global__ void calculVoltDStep2bis(double* VoltageRealIm, double* RMGgrid, double* RPGgrid, float* CoresVoiLin, float* CoresBusLin, float* nLines, float* CoresTrans, int B, int BL2) {

    int thIdx = threadIdx.x;
    //int i = blockIdx.x; un seul bloc
    int size = blockDim.x;
    

    extern __shared__ double RID[];
    double* Voltage = &RID[2 * BL2];
    for (int l = thIdx; l < BL2; l += size) { // coalecent
        RID[l] = RMGgrid[l];
        RID[l + BL2] = RPGgrid[l];
    }
    for (int k = thIdx; k < 2 * B; k+=size) {
        Voltage[k] = VoltageRealIm[k];
    }
    __syncthreads();

    for (int iter = 0; iter < B - 1; iter++) {
        int begin = CoresBusLin[iter]; // k = CoresBusLin[iter]; !!!
        int end = begin + nLines[iter]; // k + nLines[iter]

        double ei = Voltage[iter];
        double fi = Voltage[iter + B];

        for (int l = thIdx + begin + 1; l < end; l += size) { // voisin
            int j = CoresVoiLin[l];

            if (j > iter) {
                int lTrans = CoresTrans[l];

                double ri = RID[lTrans]; // acc�s pas du tout coalescent mais c'est sur la m�moire partag�
                double li = RID[lTrans + BL2];


                double db1 = ri * ei - li * fi;
                double db2 = li * ei + ri * fi;

                Voltage[j] = Voltage[j] - db1;
                Voltage[j + B] = Voltage[j + B] - db2;
            }
        }
        __syncthreads();
    }

    for (int k = thIdx; k < 2 * B; k += size) {
        VoltageRealIm[k] = Voltage[k];
    }
    __syncthreads();
}



__global__ void calcWinterCarD(double* Pinter, double* Qinter, double* E, double* Glin, double* Blin, float* CoresVoiLin, float* CoresBusLin, float* nLines, int B) {


    int index = threadIdx.x;
    int step = blockDim.x;
    int i = blockIdx.x;
    extern __shared__ double shED[];
    int begin = CoresBusLin[i];
    int end = begin + nLines[blockIdx.x];
    int B2 = 2 * B;

    for (int n = index; n < B2; n += step)
    {
        shED[n] = E[n];
    }
    __syncthreads();

    for (int l = begin + index; l < end; l += step) {
        int k = CoresVoiLin[l];
        double g = Glin[l];
        double b = Blin[l];
        double a = g * shED[k] - b * shED[k + B];
        double c = b * shED[k] + g * shED[k + B];

        Pinter[l] = shED[i] * a + shED[i + B] * c;
        Qinter[l] = shED[i + B] * a - shED[i] * c;

    }

}




__global__ void calcEGPUD(double* ED, double* VoltageRealImD, int B) {

    int thIdx = threadIdx.x + blockDim.x * blockIdx.x;
    int size = gridDim.x * blockDim.x;

    for (int i = thIdx; i < B; i += size) {
        double Rev = VoltageRealImD[i];
        double Imv = VoltageRealImD[i + B];


        ED[i + B] = sqrt(Rev * Rev + Imv * Imv);
        ED[i] = atan2(Imv, Rev);
    }

}
